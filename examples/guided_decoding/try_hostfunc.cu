#include <cstdio>
#include <hip/hip_runtime.h>

// Example host function
void myHostCallback(void* userData)
{
    printf("Host callback executed: %s\n", static_cast<char*>(userData));
}

int main()
{
    hipStream_t stream;
    hipGraph_t graph;
    hipGraphExec_t graphExec;
    char message[] = "Hello 0 from host node!";

    hipStreamCreate(&stream);

    // Begin CUDA stream capture
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);

    // [Device work would be enqueued here, e.g. kernel launches, memcpys]

    // Schedule host function as a host node in the graph
    hipLaunchHostFunc(stream, myHostCallback, message);

    // End capture; the graph now includes the host node
    hipStreamEndCapture(stream, &graph);

    // Instantiate executable graph
    hipGraphInstantiate(&graphExec, graph, nullptr, nullptr, 0);

    // Launch the graph; 'myHostCallback' will execute as a host node
    for (int i = 0; i < 10; i++)
    {
        message[6] = '0' + i;
        hipGraphLaunch(graphExec, stream);
        hipStreamSynchronize(stream);
    }

    // Cleanup
    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);
    hipStreamDestroy(stream);

    return 0;
}
