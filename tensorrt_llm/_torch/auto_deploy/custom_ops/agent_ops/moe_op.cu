#include <algorithm>
#include <hipblaslt.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

using bf16 = __hip_bfloat16;

__global__ void build_expert_maps_kernel(int const* __restrict__ selected_experts,
    float const* __restrict__ routing_weights, int const* __restrict__ expert_offsets,
    int* __restrict__ expert_write_counters, int* __restrict__ token_indices, float* __restrict__ routing_gathered,
    int batch_size, int num_selected, int num_experts)
{
    int token = blockIdx.x * blockDim.x + threadIdx.x;

    if (token < batch_size)
    {
#pragma unroll
        for (int s = 0; s < num_selected; ++s)
        {
            int expert = selected_experts[token * num_selected + s];
            if (expert >= 0 && expert < num_experts)
            {
                int base = expert_offsets[expert];
                int pos = atomicAdd(&expert_write_counters[expert], 1);
                int write_pos = base + pos;

                token_indices[write_pos] = token;
                routing_gathered[write_pos] = routing_weights[token * num_selected + s];
            }
        }
    }
}

__global__ void gather_features_kernel(bf16 const* __restrict__ x, int const* __restrict__ token_indices,
    bf16* __restrict__ x_gathered, int start_idx, int num_tokens, int hidden_dim, int batch_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = num_tokens * hidden_dim;

    for (int i = idx; i < total; i += blockDim.x * gridDim.x)
    {
        int local_token = i / hidden_dim;
        int feat = i % hidden_dim;
        int orig_token = token_indices[start_idx + local_token];

        if (orig_token >= 0 && orig_token < batch_size && feat < hidden_dim)
        {
            x_gathered[i] = x[orig_token * hidden_dim + feat];
        }
        else
        {
            x_gathered[i] = __float2bfloat16(0.0f);
        }
    }
}

__global__ void relu_squared_kernel(bf16* data, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = idx; i < size; i += blockDim.x * gridDim.x)
    {
        float val = __bfloat162float(data[i]);
        val = fmaxf(val, 0.0f);
        data[i] = __float2bfloat16(val * val);
    }
}

__global__ void scatter_output_kernel(bf16 const* __restrict__ expert_out, int const* __restrict__ token_indices,
    float const* __restrict__ routing_weights, bf16* __restrict__ final_output, int start_idx, int num_tokens,
    int hidden_dim, int batch_size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total = num_tokens * hidden_dim;

    for (int i = idx; i < total; i += blockDim.x * gridDim.x)
    {
        int local_token = i / hidden_dim;
        int feat = i % hidden_dim;
        int orig_token = token_indices[start_idx + local_token];

        if (orig_token >= 0 && orig_token < batch_size && feat < hidden_dim)
        {
            float weight = routing_weights[start_idx + local_token];
            float val = __bfloat162float(expert_out[i]) * weight;
            bf16 val_bf16 = __float2bfloat16(val);
            atomicAdd(&final_output[orig_token * hidden_dim + feat], val_bf16);
        }
    }
}

__global__ void count_expert_tokens_kernel(int const* __restrict__ selected_experts, int* __restrict__ expert_counts,
    int batch_size, int num_selected, int num_experts)
{
    extern __shared__ int smem_counts[];

    for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
    {
        smem_counts[e] = 0;
    }
    __syncthreads();

    for (int token = blockIdx.x * blockDim.x + threadIdx.x; token < batch_size; token += gridDim.x * blockDim.x)
    {
#pragma unroll
        for (int s = 0; s < num_selected; ++s)
        {
            int expert = selected_experts[token * num_selected + s];
            if (expert >= 0 && expert < num_experts)
            {
                atomicAdd(&smem_counts[expert], 1);
            }
        }
    }
    __syncthreads();

    for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
    {
        if (smem_counts[e] > 0)
        {
            atomicAdd(&expert_counts[e], smem_counts[e]);
        }
    }
}

__global__ void prefix_sum_kernel(int const* __restrict__ counts, int* __restrict__ offsets, int num_experts)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        offsets[0] = 0;
        for (int i = 0; i < num_experts; ++i)
        {
            offsets[i + 1] = offsets[i] + counts[i];
        }
    }
}

__global__ void zero_counters_kernel(int* __restrict__ counters, int num_experts)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_experts)
    {
        counters[idx] = 0;
    }
}

// Note: Removed the duplicate cublasLtGetStatusString definition to fix compilation.
// Use cublasLtGetStatusString from cublasLt.h directly.

void launch_gpu_implementation(void* x, void* selected_experts, void* routing_weights, void** w1_weights,
    void** w2_weights, void* output, int batch_size, int hidden_dim, int intermediate_dim, int num_experts,
    int num_selected, hipStream_t stream)
{
    int threads = 256;
    hipMemsetAsync(output, 0, batch_size * hidden_dim * sizeof(bf16), stream);

    int* expert_counts;
    int* expert_offsets;
    int* expert_write_counters;
    hipMallocAsync(&expert_counts, num_experts * sizeof(int), stream);
    hipMallocAsync(&expert_offsets, (num_experts + 1) * sizeof(int), stream);
    hipMallocAsync(&expert_write_counters, num_experts * sizeof(int), stream);

    hipMemsetAsync(expert_counts, 0, num_experts * sizeof(int), stream);

    int blocks = (batch_size + threads - 1) / threads;
    size_t smem_size = num_experts * sizeof(int);
    count_expert_tokens_kernel<<<blocks, threads, smem_size, stream>>>(
        (int const*) selected_experts, expert_counts, batch_size, num_selected, num_experts);

    prefix_sum_kernel<<<1, 1, 0, stream>>>(expert_counts, expert_offsets, num_experts);

    blocks = (num_experts + threads - 1) / threads;
    zero_counters_kernel<<<blocks, threads, 0, stream>>>(expert_write_counters, num_experts);

    int total_assignments = batch_size * num_selected;
    int* token_indices_all;
    float* routing_gathered_all;
    hipMallocAsync(&token_indices_all, total_assignments * sizeof(int), stream);
    hipMallocAsync(&routing_gathered_all, total_assignments * sizeof(float), stream);

    blocks = (batch_size + threads - 1) / threads;
    build_expert_maps_kernel<<<blocks, threads, 0, stream>>>((int const*) selected_experts,
        (float const*) routing_weights, expert_offsets, expert_write_counters, token_indices_all, routing_gathered_all,
        batch_size, num_selected, num_experts);

    std::vector<int> h_counts(num_experts);
    std::vector<int> h_offsets(num_experts + 1);
    hipMemcpyAsync(h_counts.data(), expert_counts, num_experts * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipMemcpyAsync(h_offsets.data(), expert_offsets, (num_experts + 1) * sizeof(int), hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    hipblasLtHandle_t handle;
    hipblasLtCreate(&handle);

    int max_tokens = 0;
    for (int i = 0; i < num_experts; ++i)
    {
        if (h_counts[i] > max_tokens)
            max_tokens = h_counts[i];
    }
    max_tokens += 1;

    bf16* x_gathered;
    bf16* intermediate;
    bf16* expert_out;
    hipMallocAsync(&x_gathered, max_tokens * hidden_dim * sizeof(bf16), stream);
    hipMallocAsync(&intermediate, max_tokens * intermediate_dim * sizeof(bf16), stream);
    hipMallocAsync(&expert_out, max_tokens * hidden_dim * sizeof(bf16), stream);

    for (int expert_idx = 0; expert_idx < num_experts; ++expert_idx)
    {
        int num_tokens = h_counts[expert_idx];
        int offset = h_offsets[expert_idx];

        if (num_tokens == 0)
            continue;

        blocks = (num_tokens * hidden_dim + threads - 1) / threads;
        gather_features_kernel<<<blocks, threads, 0, stream>>>(
            (bf16 const*) x, token_indices_all, x_gathered, offset, num_tokens, hidden_dim, batch_size);

        hipblasLtMatmulDesc_t matmulDesc;
        hipblasLtMatmulDescCreate(&matmulDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);

        {
            hipblasOperation_t opA = HIPBLAS_OP_T;
            hipblasOperation_t opB = HIPBLAS_OP_N;
            hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opA, sizeof(opA));
            hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opB, sizeof(opB));

            hipblasLtMatrixLayout_t Aop_desc, Bop_desc, Cop_desc;
            hipblasLtMatrixLayoutCreate(&Aop_desc, HIP_R_16BF, hidden_dim, intermediate_dim, hidden_dim);
            hipblasLtMatrixLayoutCreate(&Bop_desc, HIP_R_16BF, hidden_dim, num_tokens, hidden_dim);
            hipblasLtMatrixLayoutCreate(&Cop_desc, HIP_R_16BF, intermediate_dim, num_tokens, intermediate_dim);

            float alpha = 1.0f, beta = 0.0f;

            hipblasStatus_t stat1
                = hipblasLtMatmul(handle, matmulDesc, &alpha, w1_weights[expert_idx], Aop_desc, x_gathered, Bop_desc,
                    &beta, intermediate, Cop_desc, intermediate, Cop_desc, nullptr, nullptr, 0, stream);
            if (stat1 != HIPBLAS_STATUS_SUCCESS)
            {
                std::cerr << "hipblasLtMatmul (first GEMM) failed: " << cublasLtGetStatusString(stat1) << std::endl;
            }

            hipblasLtMatrixLayoutDestroy(Aop_desc);
            hipblasLtMatrixLayoutDestroy(Bop_desc);
            hipblasLtMatrixLayoutDestroy(Cop_desc);
        }

        {
            int elems = num_tokens * intermediate_dim;
            int act_blocks = (elems + threads - 1) / threads;
            relu_squared_kernel<<<act_blocks, threads, 0, stream>>>(intermediate, elems);
        }

        {
            hipblasOperation_t opA = HIPBLAS_OP_T;
            hipblasOperation_t opB = HIPBLAS_OP_N;
            hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &opA, sizeof(opA));
            hipblasLtMatmulDescSetAttribute(matmulDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &opB, sizeof(opB));

            hipblasLtMatrixLayout_t Aop_desc2, Bop_desc2, Cop_desc2;
            hipblasLtMatrixLayoutCreate(&Aop_desc2, HIP_R_16BF, intermediate_dim, hidden_dim, intermediate_dim);
            hipblasLtMatrixLayoutCreate(&Bop_desc2, HIP_R_16BF, intermediate_dim, num_tokens, intermediate_dim);
            hipblasLtMatrixLayoutCreate(&Cop_desc2, HIP_R_16BF, hidden_dim, num_tokens, hidden_dim);

            float alpha = 1.0f, beta = 0.0f;

            hipblasStatus_t stat2
                = hipblasLtMatmul(handle, matmulDesc, &alpha, w2_weights[expert_idx], Aop_desc2, intermediate, Bop_desc2,
                    &beta, expert_out, Cop_desc2, expert_out, Cop_desc2, nullptr, nullptr, 0, stream);
            if (stat2 != HIPBLAS_STATUS_SUCCESS)
            {
                std::cerr << "hipblasLtMatmul (second GEMM) failed: " << cublasLtGetStatusString(stat2) << std::endl;
            }

            hipblasLtMatrixLayoutDestroy(Aop_desc2);
            hipblasLtMatrixLayoutDestroy(Bop_desc2);
            hipblasLtMatrixLayoutDestroy(Cop_desc2);
        }

        blocks = (num_tokens * hidden_dim + threads - 1) / threads;
        scatter_output_kernel<<<blocks, threads, 0, stream>>>(expert_out, token_indices_all, routing_gathered_all,
            (bf16*) output, offset, num_tokens, hidden_dim, batch_size);

        hipblasLtMatmulDescDestroy(matmulDesc);
    }

    hipFreeAsync(expert_counts, stream);
    hipFreeAsync(expert_offsets, stream);
    hipFreeAsync(expert_write_counters, stream);
    hipFreeAsync(token_indices_all, stream);
    hipFreeAsync(routing_gathered_all, stream);
    hipFreeAsync(x_gathered, stream);
    hipFreeAsync(intermediate, stream);
    hipFreeAsync(expert_out, stream);

    hipblasLtDestroy(handle);
}
