#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: NVIDIA TensorRT Source Code License Agreement
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "cuda_hint.cuh"
#include "defines.h"
#include "ldgsts.cuh"
#include "mha.h"
#include "mhaUtils.cuh"
#include "mma.cuh"
#include "utils.cuh"

#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#ifndef GENERATE_CUBIN
#include "hostUtils.h"
#include <hip/hip_runtime.h>
#ifndef NDEBUG
#include <cstdio>
#endif
#endif

// There are 4 ways to pass ctaRowMax backward from gemm1 warps to gemm0 warps:
//  1. Protect with xFwdBarriers+xBwdBarriers. This way, ctaRowMax is available to gemm0 warps together with x tiles and
//  warpRowMax/warpRowSum. But ctaRowMax is required before warp tile online softmax, while the other buffers is needed
//  only after online softmax. So xBwdBarriers wait will need to be moved before online softmax.
//  2. Similar to approach 1, but we add an additional register copy of ctaRowMax in gemm0 warps. It's loaded from smem
//  ctaRowMax after warp tile online softmax, so the current warp tile can't use it. But we can pass it to next
//  iteration so softmax of next tile can use it. The update will be delayed by 1 more iteration and we need one or two
//  more registers. Alternatively, put the extra copy in shared memory, so we have double buffer for ctaRowMax.
//  3. Protected with dedicated backward barriers (xFwdBarriers + ctaRowmaxBwdBarriers). Then we don't have drawbacks of
//  1 or 2, but we need extra smem barriers and extra arrive/wait instructions.
//  4. No protection, just use volatile read/write. This approach gives most timely update and has lowest cost, but the
//  result is non-deterministic up to an small numeric error.
// #define CTA_ROW_MAX_BACKWARD_METHOD 4
// 1 is 8% slower than 4. 2/3 are 10% slower than 4.
#define CTA_ROW_MAX_BACKWARD_METHOD 1

static_assert(inputElemSize >= cacheElemSize);

constexpr uint32_t cacheElemsPerGrain = exactDiv(grainBytes, cacheElemSize);
constexpr uint32_t inputElemsPerGrain = exactDiv(grainBytes, inputElemSize);
constexpr bool enableMicroFastPath = false;

// x: horizontal stacking for cta horizontal tile size
// y: vertical stacking for cta vertical tile size
// z: must be 2 for warp specialization.
constexpr uint3 ctaShapeInWarps = {4, 1, 2};

static_assert(ctaShapeInWarps.z == 2); // for warp specialization
constexpr uint32_t nbWarpsPerCta = ctaShapeInWarps.x * ctaShapeInWarps.y * ctaShapeInWarps.z;
constexpr uint32_t ctaSize = warp_size * nbWarpsPerCta;

#if SPEC_DEC
// Use 32 row size
constexpr uint32_t nbValidRows = rowsPerBlock;
static_assert(nbValidRows <= 32u);
#else
constexpr uint32_t nbValidRows = headGrpSize * beamWidth;
#endif
constexpr uint2 warpTile = {64, roundUp(nbValidRows, 16U)};
static_assert(nbValidRows <= warpTile.y);

constexpr uint32_t gemm1WarpsPerGrp = exactDiv(headElems, warpTile.x);
constexpr uint32_t gemm1NbWarpGrps
    = exactDiv(ctaShapeInWarps.x, gemm1WarpsPerGrp);       // warp groups split along seqLen dim.

constexpr uint2 ctaTile = {warpTile.x * ctaShapeInWarps.x, // if .x is greater than headSize, then gemm1 uses split-K
    warpTile.y* ctaShapeInWarps.y};

constexpr uint32_t cvtExpansion = exactDiv(inputElemSize, cacheElemSize);

#ifndef __CUDA_ARCH__
constexpr uint32_t preferedKHeadPartBytes = 64;
__constant__ constexpr uint32_t cacheVTileSeqLen = 32;
#else
#if __CUDA_ARCH__ == 860 || __CUDA_ARCH__ == 890 || __CUDA_ARCH__ == 1200
constexpr uint32_t preferedKHeadPartBytes = 64;
__constant__ constexpr uint32_t cacheVTileSeqLen = 32;
#elif __CUDA_ARCH__ == 800 || __CUDA_ARCH__ == 870 || __CUDA_ARCH__ == 900
constexpr uint32_t preferedKHeadPartBytes = 128;
__constant__ constexpr uint32_t cacheVTileSeqLen = 64;
#else
#error "perferedKHeadPartBytes not defined"
#endif
#endif
constexpr uint32_t kHeadPartBytes = mha::min(preferedKHeadPartBytes, paddedCacheHeadBytes);
// constexpr uint32_t cacheElemsPerKHeadPart = exactDiv(kHeadPartBytes, cacheElemSize);

constexpr bool persistentQ = paddedInputHeadBytes * ctaTile.y <= (16u << 10);
static_assert(persistentQ);
constexpr uint32_t qHeadPartBytes = persistentQ ? paddedInputHeadBytes : kHeadPartBytes;
constexpr uint32_t qHeadPartElems = exactDiv(qHeadPartBytes, inputElemSize);

constexpr uint32_t nbPartsPerCacheKHead = exactDiv(paddedCacheHeadBytes, kHeadPartBytes);
constexpr uint32_t nbPartsPerInputKHead = exactDiv(paddedInputHeadBytes, kHeadPartBytes);
constexpr uint32_t nbPartsPerInputQHead = exactDiv(paddedInputHeadBytes, qHeadPartBytes);

// false - each warp load V tiles independent of each other; true - all warps in a warp group load V tiles together.
// @fixme: when true, and nbVBuffers is only 2, we need to sync all warps in a group after finishing using a buffer and
// before refill it with prefetch data. We may need at least 3.
constexpr bool grpLoadV = GRP_LOAD_V;

// number of shared memory buffers for latency hiding
constexpr uint32_t nbQBuffers = mha::min(nbPartsPerInputQHead, 2u); // for latency hiding
constexpr uint32_t nbKBuffers = 2;                                  // for latency hiding
constexpr uint32_t nbVBuffers = 2; // @fixme: H100 SXM need more in-flight requests. may need to increase this.
constexpr uint32_t nbXBuffers = 1;

__device__ inline uint3 getWarpIdx(Warp const& warp = this_warp())
{
    return uint3{ctaShapeInWarps.x == 1 ? 0 : makeWarpUniform(warp, threadIdx.x / warp_size),
        ctaShapeInWarps.y == 1 ? 0 : makeWarpUniform(warp, threadIdx.y),
        ctaShapeInWarps.z == 1 ? 0 : makeWarpUniform(warp, threadIdx.z)};
}

__device__ inline uint32_t gemm1WarpGrpIdx(uint32_t warpIdxX)
{
    return gemm1NbWarpGrps == 1 ? 0 : warpIdxX / gemm1WarpsPerGrp;
}

__device__ inline uint32_t gemm1WarpIdxInGrp(uint32_t warpIdxX)
{
    return gemm1WarpsPerGrp == 1 ? 0 : (gemm1NbWarpGrps == 1 ? warpIdxX : warpIdxX % gemm1WarpsPerGrp);
}

constexpr uint32_t instM = 16;
constexpr uint32_t instN = 8;
// constexpr uint32_t instK = 16;
constexpr uint32_t quadPerWarp = warp_size / 4;

using QuadRegRowMax
    = Vec<float, divUp(warpTile.y, warp_size) * 4>;             // data is replicated across 4 threads in a MMA quad.
using ThrdRegRowMax = Vec<float, divUp(warpTile.y, warp_size)>; // unlike QuadRegRowMax, not replicated.
using UniformRescaleMask = Vec<uint32_t, divUp(warpTile.y, warp_size)>; // uniform and stored in UR

__device__ inline bool any(UniformRescaleMask const& x)
{
    uint32_t val = 0U;
#pragma unroll
    for (uint32_t i = 0; i < x.size; i++)
    {
        uint32_t word = x[i];
        constexpr uint32_t wordBits = 32;
        if (warpTile.y % wordBits != 0 && i + 1 == x.size)
        {
            constexpr uint32_t validBits = warpTile.y % wordBits;
            word &= ((1U << validBits) - 1);
        }
        val |= word;
    }
    return val != 0;
}

#ifndef NDEBUG
__device__ inline void printRowMax(ThrdRegRowMax const& src)
{
    for (uint32_t i = 0; i < warp_size * src.size; i++)
    {
        if (laneId() == i % warp_size)
        {
            printf("%f%s", src[i / warp_size], i == 31 ? "\n" : " ");
        }
        __syncwarp();
    }
}

__device__ inline void printRowMax(QuadRegRowMax const& src)
{
    for (uint32_t i = 0; i < src.size / 4; i++)
    {
        for (uint32_t j = 0; j < 8; j++)
        {
            if (laneId() == 4 * j)
            {
                for (uint32_t k = 0; k < 4; k++)
                {
                    printf("%f%s", src[i * 4 + k], i == 31 ? "\n" : " ");
                }
            }
            __syncwarp();
        }
    }
}
#endif

struct alignas(16) SMemWarpRowMax
{
    __device__ inline float const& operator[](uint32_t idxRow) const
    {
        assert(idxRow < ThrdRegRowMax::size * warp_size);
        uint32_t const idxInstM8 = idxRow / quadPerWarp;
        return data[ThrdRegRowMax::size == 1 ? 0 : idxInstM8 / 4][idxRow % quadPerWarp][idxInstM8 % 4];
    }

    __device__ inline float& operator[](uint32_t idxRow)
    {
        return const_cast<float&>(static_cast<SMemWarpRowMax const&>(*this)[idxRow]);
    }

    // When data is register, data is replicate across 4 threads in a quad.
    template <bool asVolatile>
    __device__ inline QuadRegRowMax const loadToRegForQuad(Warp const& warp) const
    {
        uint32_t const idxQuad = laneId() / 4;
        QuadRegRowMax result;
#pragma unroll
        for (uint32_t i = 0; i < divUp(warpTile.y, quadPerWarp * 4); i++)
        {
            auto const& src = data[i][idxQuad];
            auto& dst = reinterpret_cast<float(&)[4]>(result[4 * i]);
            if constexpr (asVolatile)
            {
                asm volatile("ld.volatile.shared.v4.f32 {%0, %1, %2, %3}, [%4];\n"
                             : "=f"(dst[0]), "=f"(dst[1]), "=f"(dst[2]), "=f"(dst[3])
                             : "l"(__cvta_generic_to_shared(&src)));
            }
            else
            {
                reinterpret_cast<float4&>(dst) = reinterpret_cast<float4 const&>(src);
            }
        }
        return result;
    }

    template <bool asVolatile>
    __device__ inline ThrdRegRowMax const loadToReg(Warp const& warp) const
    {
        ThrdRegRowMax result;
#pragma unroll
        for (uint32_t i = 0; i < result.size; i++)
        {
            auto const& src = this->operator[](warp_size * i + laneId());
            float& dst = result[i];
            if constexpr (asVolatile)
            {
                dst = static_cast<float const volatile&>(src);
                // asm volatile("ld.volatile.shared.f32 %0, [%1];\n"
                //     : "=f"(dst) : "l"(__cvta_generic_to_shared(&src)));
            }
            else
            {
                dst = src;
            }
        }
        return result;
    }

    template <bool asVolatile>
    __device__ inline void storeFromReg(Warp const& warp, QuadRegRowMax const& regData)
    {
        for (uint32_t i = 0; i < regData.size; i++)
        {
            assert(regData[i] == __shfl_sync(0xFU << (laneId() / 4 * 4), regData[i], 0, 4));
        }
        if (laneId() % 4 != 0)
        {
            return;
        }
        uint32_t const idxQuad = laneId() / 4;
#pragma unroll
        for (uint32_t i = 0; i < ThrdRegRowMax::size; i++)
        {
            auto& dst = data[i][idxQuad];
            auto const& src = reinterpret_cast<float const(&)[4]>(regData[4 * i]);
            if constexpr (asVolatile)
            {
                asm volatile(
                    "st.volatile.shared.v4.f32 [%0], {%1, %2, %3, %4};\n" ::"l"(__cvta_generic_to_shared(&dst)),
                    "f"(src[0]), "f"(src[1]), "f"(src[2]), "f"(src[3]));
            }
            else
            {
                reinterpret_cast<float4&>(dst) = reinterpret_cast<float4 const&>(src);
            }
        }
    }

    template <bool asVolatile>
    __device__ inline void storeFromReg(Warp const& warp, ThrdRegRowMax const& regData)
    {
#pragma unroll
        for (uint32_t i = 0; i < ThrdRegRowMax::size; i++)
        {
            auto& dst = this->operator[](warp_size * i + laneId());
            assert(!hasBankConflict(&dst));
            float const src = regData[i];
            if constexpr (asVolatile)
            {
                static_cast<float volatile&>(dst) = src;
            }
            else
            {
                dst = src;
            }
        }
    }

    __device__ inline void atomicMaxUpdate(Warp const& warp, ThrdRegRowMax const& regData)
    {
#pragma unroll
        for (uint32_t i = 0; i < ThrdRegRowMax::size; i++)
        {
            auto& dst = this->operator[](warp_size * i + laneId());
            assert(!hasBankConflict(&dst));
            float const src = regData[i];
            atomicMax(&dst, src);
        }
    }

    float data[ThrdRegRowMax::size][quadPerWarp][4];
};

// idxMat8 is the reduced row index in 8-row unit.
__device__ inline float replicateValForQuad(Warp const& warp, ThrdRegRowMax const& src, uint32_t idxMat8)
{
    assertWarpConverged();
    uint32_t const i = idxMat8 / 4;
    uint32_t const j = idxMat8 % 4;
    return __shfl_sync(~0U, src[i], quadPerWarp * j + laneId() / 4);
}

__device__ inline QuadRegRowMax replicateForQuad(Warp const& warp, ThrdRegRowMax const& src)
{
    assertWarpConverged();
    QuadRegRowMax dst;
#pragma unroll
    for (uint32_t i = 0; i < src.size; i++)
    {
#pragma unroll
        for (uint32_t j = 0; j < 4; j++)
        {
            dst[i * 4 + j] = __shfl_sync(~0U, src[i], quadPerWarp * j + laneId() / 4);
            assert(dst[i * 4 + j] == replicateValForQuad(warp, src, i * 4 + j));
        }
    }
    return dst;
}

// cacheVTileSeqLen may be smaller than x cols, so we need multiple v tiles per X tile.
constexpr uint32_t nbCacheVTilesPerXTile = exactDiv(warpTile.x, cacheVTileSeqLen);

constexpr uint32_t nbWarpGrpsPerXTile = mha::min(nbCacheVTilesPerXTile, gemm1NbWarpGrps);

#if USE_PAGED_KV_CACHE
constexpr uint32_t nbPagesPerWarpTile = (warpTile.x <= tokensPerPage ? 1U : exactDiv(warpTile.x, tokensPerPage));
using KCachePageIndices = Vec<KVCachePageIndex, nbPagesPerWarpTile>;
constexpr uint32_t nbPagesPerVTile
    = (cacheVTileSeqLen <= tokensPerPage ? 1 : exactDiv(cacheVTileSeqLen, tokensPerPage));
using VCachePageIndices = Vec<KVCachePageIndex, nbPagesPerVTile>;
#endif

static_assert(ctaShapeInWarps.y == 1);

struct alignas(128) SharedMem
{
    using QSmemBuffer = Array2D<LdGrain, warpTile.y, exactDiv(qHeadPartBytes, grainBytes)>;
    using KSmemBuffer = Array2D<LdGrain, warpTile.x, exactDiv(kHeadPartBytes, grainBytes)>;
    using XSmemBuffer = Array2D<LdGrain, warpTile.y, exactDiv(inputElemSize* warpTile.x, grainBytes)>;
    using VSmemBuffer
        = Array2D<LdGrain, cacheVTileSeqLen, exactDiv(grpLoadV ? headElems : warpTile.x, cacheElemsPerGrain)>;

    QSmemBuffer q[ctaShapeInWarps.y][nbQBuffers];
    KSmemBuffer k[ctaShapeInWarps.x][nbKBuffers];
    XSmemBuffer x[ctaShapeInWarps.y][ctaShapeInWarps.x];
    static_assert(nbXBuffers == 1);
    VSmemBuffer v[gemm1NbWarpGrps][grpLoadV ? 1 : gemm1WarpsPerGrp][nbVBuffers];

    SMemWarpRowMax warpRowMax[ctaShapeInWarps.y][ctaShapeInWarps.x]; // the max used when computing this->x
    SMemWarpRowMax warpRowSum[ctaShapeInWarps.y][ctaShapeInWarps.x]; // the row sum of gemm0 output

#if CTA_ROW_MAX_BACKWARD_METHOD == 1 || CTA_ROW_MAX_BACKWARD_METHOD == 2 || CTA_ROW_MAX_BACKWARD_METHOD == 3
    // protected with xFwdBarriers+xBwdBarriers for CTA_ROW_MAX_BACKWARD_METHOD 1 or 2, and with
    // xFwdBarriers+ctaRowMaxBwdBarriers for 3. Cannot reuse warpRowMax because a gemm1 warp is not sure whether other
    // gemm1 warps have finished using it, unless we want to pay extra sync.
    SMemWarpRowMax ctaRowMax[ctaShapeInWarps.y][ctaShapeInWarps.x];
#elif CTA_ROW_MAX_BACKWARD_METHOD == 4
    SMemWarpRowMax ctaRowMax[ctaShapeInWarps.y]; // just a hint, no strict protection required if you don't care about
                                                 // non-deterministic output (up to a small numeric error)
#endif

#if BEAM_WIDTH > 1
    Vec<uint32_t, warpTile.x> gemm0CacheIndir[ctaShapeInWarps.x];
    Vec<uint32_t, cacheVTileSeqLen> gemm1CacheIndir[grpLoadV ? gemm1NbWarpGrps : ctaShapeInWarps.x];
#if USE_PAGED_KV_CACHE
    Vec<KCachePageIndices, beamWidth> kCachePages[ctaShapeInWarps.x];
    Vec<VCachePageIndices, beamWidth> vCachePages[grpLoadV ? gemm1NbWarpGrps : ctaShapeInWarps.x];
#endif
#endif

    using Barrier = CtaBarrier;

    Barrier qBarrier[ctaShapeInWarps.y];
    // Beside X buffers, also protects warpRowMax and warpRowSum. For CTA_ROW_MAX_BACKWARD_METHOD==1 or 2, also
    // ctaRowMax.
    CtaBarrierPair xBarriers[ctaShapeInWarps.y][ctaShapeInWarps.x];
#if CTA_ROW_MAX_BACKWARD_METHOD == 3
    Barrier ctaRowMaxBwdBarriers[ctaShapeInWarps.y]
                                [ctaShapeInWarps.x]; // xFwdBarriers+ctaRowMaxBwdBarriers protects ctaRowMax
#endif

#if GRP_LOAD_V
    static constexpr uint32_t nbOtherBarriers = nbVBuffers * gemm1NbWarpGrps + gemm1NbWarpGrps;
    Barrier otherBarriers[nbOtherBarriers];
#endif
    __device__ inline Barrier* vBarrier(uint32_t warpGrpIdx, uint32_t idxBuf)
    {
#if GRP_LOAD_V
        return &reinterpret_cast<Barrier(&)[gemm1NbWarpGrps][nbVBuffers]>(otherBarriers)[warpGrpIdx][idxBuf];
#else
        return nullptr;
#endif
    }

    __device__ inline Barrier* warpGrpBar(uint32_t warpGrpIdx)
    {
#if GRP_LOAD_V
        return &otherBarriers[nbVBuffers * gemm1NbWarpGrps + warpGrpIdx];
#else
        return nullptr;
#endif
    }
};

CUBIN_EXPORT __device__ constexpr uint32_t smemSize = sizeof(SharedMem);
#ifdef __CUDA_ARCH__
static_assert(smemSize < kMAX_SMEM_SIZE);
#endif

#if 0
template <bool swizzled, uint32_t rows, uint32_t cols>
__device__ inline void smemRotateInplace(Warp const& Warp, Array2D<LdGrain, rows, cols>& data, uint32_t idxPart, uint32_t idxToken) {
    static_assert(inputSeqLen == 1);
    constexpr uint32_t rowElems = inputElemsPerGrain * cols;
    constexpr uint32_t nbParts = exactDiv(headElems, idxPart);
    static_assert(nbParts % 2 == 0);
    bool const isFirstHalf = (idxPart < nbParts / 2);
    static_assert(mha::is_same_v<InputElem, half>, "not implemented");
    if constexpr (cols <= warp_size) {
        static_assert(warp_size % cols == 0);
        constexpr uint32_t thrdGrpSize = LdGrain::size * cols;
        uint32_t const idxThrdGrp = laneId() / thrdGrpSize;
        uint32_t const thrdGrpLane = laneId() % thrdGrpSize;
        constexpr uint32_t nbThrdGrps = warp_size / thrdGrpSize;
        static_assert(warp_size % thrdGrpSize == 0);
        constexpr uint32_t nbElemsPerWord = exactDiv(sizeof(LdGrain::Elem), inputElemSize);
        Vec<float, nbElemsPerWord> cosAngles;
        Vec<float, nbElemsPerWord> sinAngles;
#pragma unroll
        for (uint32_t i = 0; i < angles.size; i++) {
            uint32_t const n = rowElems * (idxPart % (nbParts / 2)) + angles.size * thrdGrpLane + i;
            float const angle = powf(1E-4f, n * (2.f / headElems)) * idxToken;
            sincosf(angle, &sinAngles[i], &cosAngles[i]);
        }

        constexpr uint32_t nbIters = exactDiv(rows, nbThrdGrps);
#pragma unroll
        for (uint32_t i = 0; i < nbIters; i++) {
            auto const word = data.template at<swizzled>(nbThrdGrps * i + idxThrdGrp, thrdGrpLane / LdGrain::size)[thrdGrpLane % LdGrain::size];
            float2 const val = __half22float2(reinterpret_cast<InputElem2 const&>(word));
            Vec<float, nbElemsPerWord> result;
#pragma unroll
            for (uint32_t j = 0; j < nbElemsPerWord; j++) {
                if (isFirstHalf) {
                    result[j] = cosAngles[j] * ;
                }
            }
        }
    }
    else {
        static_assert(cols <= warp_size, "not implemented");
    }
}
#endif

using InstAcc = Array2D<float, 2, 2>;
using WarpAcc = Array2D<InstAcc, exactDiv(warpTile.y, instM), exactDiv(warpTile.x, instN)>;

__device__ inline void applyMask(Warp const& warp, WarpAcc& acc, uint32_t validColBeg, uint32_t validColEnd)
{
    uint32_t const idxInQuad = laneId() % 4;
    uint32_t const idxQuad = laneId() / 4;
#pragma unroll
    for (uint32_t n = 0; n < acc.cols; n++)
    {
#pragma unroll
        for (uint32_t j = 0; j < InstAcc::cols; j++)
        {
            uint32_t const col = instN * n + InstAcc::cols * idxInQuad + j;
            if (col >= validColBeg && col < validColEnd)
            {
                continue;
            }
#pragma unroll
            for (uint32_t m = 0; m < acc.rows; m++)
            {
#pragma unroll
                for (uint32_t i = 0; i < InstAcc::rows; i++)
                {
                    acc(m, n)(i, j) = mha::numeric_limits<float>::lowest();
                }
            }
        }
    }
}

#if SPEC_DEC
#define MMAS_N_PER_MASK 2

__device__ inline void applyMaskFromInput(Warp const& warp, WarpAcc& acc, MaskType const* mask, uint32_t rowOffset,
    uint32_t nbValidCols, uint32_t qSeqLen, uint32_t actualQSeqLen, uint32_t headGrpSize)
{
    uint32_t const idxInQuad = laneId() % 4;
    uint32_t const idxQuad = laneId() / 4;
    // Packed mask is aligned with 32 bits (2 uint16_t).
    uint32_t const nbPackedMasksPerRow = divUp(qSeqLen, 32u) * 2u;
    uint16_t const* uint16Mask = reinterpret_cast<uint16_t const*>(mask);
#pragma unroll
    for (uint32_t m = 0; m < acc.rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
            uint32_t const tokenRow = min((rowOffset + instM * m + idxQuad + i * 8) / headGrpSize, actualQSeqLen - 1);
#pragma unroll
            for (uint32_t mask_n = 0; mask_n < acc.cols / MMAS_N_PER_MASK; mask_n++)
            {
                uint32_t const firstCol = instN * mask_n * MMAS_N_PER_MASK + InstAcc::cols * idxInQuad;
                uint32_t const lastCol = firstCol + instN * (MMAS_N_PER_MASK - 1) + InstAcc::cols - 1;
                uint32_t const maskPos0 = firstCol + actualQSeqLen < nbValidCols
                    ? 0u
                    : min(firstCol + actualQSeqLen - nbValidCols, actualQSeqLen - 1);
                uint32_t const maskPos1 = lastCol + actualQSeqLen < nbValidCols
                    ? 0u
                    : min(lastCol + actualQSeqLen - nbValidCols, actualQSeqLen - 1);
                uint32_t packedMask = 0u;
                uint32_t const maskPosStart = (maskPos0 / 16) * 16;
                reinterpret_cast<uint16_t*>(&packedMask)[0]
                    = uint16Mask[tokenRow * nbPackedMasksPerRow + (maskPos0 / 16)];
                reinterpret_cast<uint16_t*>(&packedMask)[1]
                    = uint16Mask[tokenRow * nbPackedMasksPerRow + (maskPos1 / 16)];
#pragma unroll
                for (uint32_t nj = 0; nj < MMAS_N_PER_MASK; nj++)
                {
#pragma unroll
                    for (uint32_t j = 0; j < InstAcc::cols; j++)
                    {
                        uint32_t const n = (mask_n * MMAS_N_PER_MASK + nj);
                        uint32_t const col = instN * n + InstAcc::cols * idxInQuad + j;
                        // bool const maskFlag = col + qSeqLen < nbValidCols ? true : mask[tokenRow * qSeqLen + (col +
                        // qSeqLen - nbValidCols)];
                        bool const maskFlag = col + actualQSeqLen < nbValidCols
                            ? true
                            : packedMask & (1u << ((col + actualQSeqLen - nbValidCols) - maskPosStart));
                        acc(m, n)(i, j) = maskFlag && col < nbValidCols ? acc(m, n)(i, j) : -INFINITY;
                    }
                }
            }
        }
    }
}
#endif

__device__ inline QuadRegRowMax warpTileOnlineSoftmax(Warp const& warp, QuadRegRowMax const& rowMaxHint, WarpAcc& acc)
{
    QuadRegRowMax rowMax = rowMaxHint;
// compute per-thread row max
#pragma unroll
    for (uint32_t n = 0; n < acc.cols; n++)
    {
#pragma unroll
        for (uint32_t j = 0; j < InstAcc::cols; j++)
        {
#pragma unroll
            for (uint32_t m = 0; m < acc.rows; m++)
            {
#pragma unroll
                for (uint32_t i = 0; i < InstAcc::rows; i++)
                {
                    rowMax[m * InstAcc::rows + i] = fmaxf(rowMax[m * InstAcc::rows + i], acc(m, n)(i, j));
                }
            }
        }
    }
// compute warp row max
#pragma unroll
    for (uint32_t xorMask = 2; xorMask != 0; xorMask /= 2)
    {
#pragma unroll
        for (uint32_t i = 0; i < rowMax.size; i++)
        {
            rowMax[i] = fmaxf(rowMax[i], __shfl_xor_sync(~0U, rowMax[i], xorMask));
        }
    }
// update acc and rowMax
#pragma unroll
    for (uint32_t m = 0; m < acc.rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
            float const maxVal = rowMax[m * InstAcc::rows + i];
            float const bias = maxVal * log2e;
#pragma unroll
            for (uint32_t n = 0; n < acc.cols; n++)
            {
#pragma unroll
                for (uint32_t j = 0; j < InstAcc::cols; j++)
                {
                    float& elem = acc(m, n)(i, j);
                    assert(maxVal >= elem);
                    elem = exp2f(elem * log2e - bias);
                }
            }
        }
    }
    return rowMax;
}

using GemmOutRegTile = Array2D<InputElem2, WarpAcc::rows * InstAcc::rows, WarpAcc::cols * exactDiv(InstAcc::cols, 2)>;

__device__ inline GemmOutRegTile toFp16(WarpAcc const& acc)
{
    GemmOutRegTile dst;
#pragma unroll
    for (uint32_t m = 0; m < acc.rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
#pragma unroll
            for (uint32_t n = 0; n < acc.cols; n++)
            {
#pragma unroll
                for (uint32_t j = 0; j < InstAcc::cols; j += 2)
                {
#if INPUT_FP16
                    dst(m * InstAcc::rows + i, (n * InstAcc::cols + j) / 2)
                        = __floats2half2_rn(acc(m, n)(i, j), acc(m, n)(i, j + 1));
#else
                    dst(m * InstAcc::rows + i, (n * InstAcc::cols + j) / 2)
                        = __floats2bfloat162_rn(acc(m, n)(i, j), acc(m, n)(i, j + 1));
#endif
                }
            }
        }
    }
    return dst;
}

__device__ inline WarpAcc toWarpAcc(GemmOutRegTile const& outTile)
{
    WarpAcc acc;
#pragma unroll
    for (uint32_t m = 0; m < acc.rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
#pragma unroll
            for (uint32_t n = 0; n < acc.cols; n++)
            {
#pragma unroll
                for (uint32_t j = 0; j < InstAcc::cols; j += 2)
                {
#if INPUT_FP16
                    float2 const fp32Vals = __half22float2(outTile(m * InstAcc::rows + i, (n * InstAcc::cols + j) / 2));
#else
                    float2 const fp32Vals
                        = __bfloat1622float2(outTile(m * InstAcc::rows + i, (n * InstAcc::cols + j) / 2));
#endif
                    acc(m, n)(i, j) = fp32Vals.x;
                    acc(m, n)(i, j + 1) = fp32Vals.y;
                }
            }
        }
    }
    return acc;
}

__device__ inline QuadRegRowMax computeRowSum(Warp const& warp, GemmOutRegTile const& src)
{
    Vec<InstAcc, exactDiv(GemmOutRegTile::rows, InstAcc::rows)> acc{};
#if INPUT_FP16
    InputElem2 const b[2][1] = {__floats2half2_rn(1, 1), __floats2half2_rn(1, 1)};
#else
    InputElem2 const b[2][1] = {__floats2bfloat162_rn(1, 1), __floats2bfloat162_rn(1, 1)};
#endif
#pragma unroll
    for (uint32_t n = 0; n < exactDiv(GemmOutRegTile::cols, 2); n++)
    {
#pragma unroll
        for (uint32_t m = 0; m < exactDiv(GemmOutRegTile::rows, 2); m++)
        {
            InputElem2 const a[2 /*kEx*/][2 /*mEx*/]
                = {src(m * 2, n * 2), src(m * 2 + 1, n * 2), src(m * 2, n * 2 + 1), src(m * 2 + 1, n * 2 + 1)};
            mma<InputElem>(acc[m].data, reinterpret_cast<uint32_t const(&)[2][2]>(a),
                reinterpret_cast<uint32_t const(&)[2][1]>(b));
        }
    }
    QuadRegRowMax rowSum;
#pragma unroll
    for (uint32_t i = 0; i < acc.size; i++)
    {
#pragma unroll
        for (uint32_t j = 0; j < InstAcc::rows; j++)
        {
            rowSum[i * InstAcc::rows + j] = acc[i](j, 0);
#pragma unroll
            for (uint32_t k = 0; k < InstAcc::cols; k++)
            {
                assert(acc[i](j, k) == acc[i](j, 0));
            }
        }
        rowSum[i * 2] = acc[i](0, 0);
        rowSum[i * 2 + 1] = acc[i](1, 0);
    }
// Sometimes there are errors in sum and they mismatch inside a quad. Force broadcast from lane 0 of each quad to
// eliminate mismatch. This has no visible impact on final result and can be removed.
#pragma unroll
    for (uint32_t i = 0; i < QuadRegRowMax::size; i++)
    {
        auto const lane0Val = __shfl_sync(0xFU << (laneId() / 4 * 4), rowSum[i], 0, 4);
        // Disable the assert, sometimes it triggers because of different orders of accumulation.
        // assert(fabs(rowSum[i] - lane0Val) < 1E-4f);
        rowSum[i] = lane0Val;
    }
    return rowSum;
}

__device__ inline void storeOrderedGemmOutTile(Warp const& warp, SharedMem::XSmemBuffer& dst, GemmOutRegTile const& src)
{
    static_assert(sizeof(dst) == sizeof(src) * warp_size);
    uint32_t const lane = laneId();
#if __CUDA_ARCH__ >= 900
    constexpr uint2 storeUnits = {4, 1}; // in 8x8 b16 matrices.
    static_assert(storeUnits.x * storeUnits.y == 4);
#pragma unroll
    for (uint32_t m = 0; m < exactDiv(dst.rows, 8 * storeUnits.y); m++)
    {
#pragma unroll
        for (uint32_t n = 0; n < exactDiv(dst.cols * grainBytes / inputElemSize, 8 * storeUnits.x); n++)
        {
            uint32_t const idxRowLocal = lane % 8;
            uint32_t const flatIdxMatLocal = lane / 8;
            uint2 const idxMatLocal = {flatIdxMatLocal % storeUnits.x, flatIdxMatLocal / storeUnits.x};
            LdGrain* const p = &dst.template at<true>(
                8 * (storeUnits.y * m + idxMatLocal.y) + idxRowLocal, storeUnits.x * n + idxMatLocal.x);

            LdGrain data;
#pragma unroll
            for (uint32_t i = 0; i < storeUnits.y; i++)
            {
#pragma unroll
                for (uint32_t j = 0; j < storeUnits.x; j++)
                {
                    data[i * storeUnits.x + j]
                        = reinterpret_cast<uint32_t const&>(src(m * storeUnits.y + i, n * storeUnits.x + j));
                }
            }
            stmatrix_4x<false>(warp, p, data);
        }
    }
#else
#pragma unroll
    for (uint32_t m = 0; m < exactDiv(dst.rows, 8); m++)
    {
#pragma unroll
        for (uint32_t n = 0; n < exactDiv(dst.cols * grainBytes / inputElemSize, 8); n++)
        {
            uint32_t const idxRowLocal = laneId() / 4;
            uint32_t const idxWordLocal = laneId() % 4;
            dst.template at<true>(8 * m + idxRowLocal, n)[idxWordLocal] = reinterpret_cast<uint32_t const&>(src(m, n));
        }
    }
#endif
}

// Reorder to compensate the reorder caused by V cache load+conversion.
__device__ inline void reorderAndStoreGemmOutTile(
    Warp const& warp, SharedMem::XSmemBuffer& dst, GemmOutRegTile const& src)
{
    static_assert(sizeof(dst) == sizeof(src) * warp_size);
    uint32_t const lane = laneId();
#pragma unroll
    for (uint32_t m = 0; m < exactDiv(dst.rows, 8); m++)
    {
#pragma unroll
        for (uint32_t n = 0; n < exactDiv(dst.cols * grainBytes / inputElemSize, 8 * 2); n++)
        {
            uint32_t const idxRowLocal = laneId() / 4;
            uint32_t const idxSegLocal = laneId() % 4;
            Vec<InputElem2, cvtExpansion> seg;
#pragma unroll
            for (uint32_t e = 0; e < cvtExpansion; e++)
            {
                seg[e] = src(m, n * cvtExpansion + e);
            }
            // reorder
            // Ideally compiler should be able to fuse this into toFp16() and just reorder input registers of F2FP
            // instructions.
            Vec<InputElem, cvtExpansion * 2> reorderedSeg;
#pragma unroll
            for (uint32_t e = 0; e < cvtExpansion; e++)
            {
                reorderedSeg[e] = seg[e].x;
                reorderedSeg[cvtExpansion + e] = seg[e].y;
            }
            static_assert(cvtExpansion <= LdGrain::size);
            constexpr uint32_t nbSegPerGrain = exactDiv(grainBytes, sizeof(seg));
            reinterpret_cast<Vec<uint32_t, cvtExpansion>&>(dst.template at<true>(8 * m + idxRowLocal,
                n * cvtExpansion + idxSegLocal / nbSegPerGrain)[idxSegLocal % nbSegPerGrain * cvtExpansion])
                = reinterpret_cast<Vec<uint32_t, cvtExpansion>&>(reorderedSeg);
        }
    }
}

__device__ inline void storeGemmOutTile(
    Warp const& warp, SharedMem::XSmemBuffer& dst, GemmOutRegTile const& src, bool reorder)
{
    if (reorder)
    {
        reorderAndStoreGemmOutTile(warp, dst, src);
    }
    else
    {
        storeOrderedGemmOutTile(warp, dst, src);
    }
}

__device__ inline GemmOutRegTile loadGemmOutTile(Warp const& warp, SharedMem::XSmemBuffer const& src)
{
    uint32_t const lane = laneId();
    GemmOutRegTile dst;
    static_assert(sizeof(src) == sizeof(dst) * warp_size);
#if __CUDA_ARCH__ >= 900
    constexpr uint2 storeUnits = {4, 1}; // in 8x8 b16 matrices.
    static_assert(storeUnits.x * storeUnits.y == 4);
#pragma unroll
    for (uint32_t m = 0; m < exactDiv(SharedMem::XSmemBuffer::rows, 8 * storeUnits.y); m++)
    {
#pragma unroll
        for (uint32_t n = 0; n < exactDiv(SharedMem::XSmemBuffer::cols * grainBytes / inputElemSize, 8 * storeUnits.x);
             n++)
        {
            uint32_t const idxRowLocal = lane % 8;
            uint32_t const flatIdxMatLocal = lane / 8;
            uint2 const idxMatLocal = {flatIdxMatLocal % storeUnits.x, flatIdxMatLocal / storeUnits.x};
            LdGrain const* const p = &src.template at<true>(
                8 * (storeUnits.y * m + idxMatLocal.y) + idxRowLocal, storeUnits.x * n + idxMatLocal.x);

            LdGrain data = ldmatrix_4x<false>(warp, p);
#pragma unroll
            for (uint32_t i = 0; i < storeUnits.y; i++)
            {
#pragma unroll
                for (uint32_t j = 0; j < storeUnits.x; j++)
                {
                    reinterpret_cast<uint32_t&>(dst(m * storeUnits.y + i, n * storeUnits.x + j))
                        = data[i * storeUnits.x + j];
                }
            }
        }
    }
#else
#pragma unroll
    for (uint32_t m = 0; m < exactDiv(SharedMem::XSmemBuffer::rows, 8); m++)
    {
#pragma unroll
        for (uint32_t n = 0; n < exactDiv(SharedMem::XSmemBuffer::cols * grainBytes / inputElemSize, 8); n++)
        {
            uint32_t const idxRowLocal = laneId() / 4;
            uint32_t const idxWordLocal = laneId() % 4;
            reinterpret_cast<uint32_t&>(dst(m, n)) = src.template at<true>(8 * m + idxRowLocal, n)[idxWordLocal];
        }
    }
#endif
    return dst;
}
// only the first nbValidRows rows are copied, to allow padding.
__device__ inline void copyOutputToGlobalMem(Warp const& warp, OutputHead* dst, uint32_t nbQHeads,
#if SPEC_DEC
    uint32_t headGrpSize, uint32_t idxHeadGrpOffset, uint32_t nbValidHeadTokens,
#else
    uint32_t idxHeadGrp,
#endif
    uint2 dstOffset, SharedMem::XSmemBuffer const& src)
{
    static_assert(sizeof(PaddedInputHead) == grainBytes * SharedMem::XSmemBuffer::cols * gemm1WarpsPerGrp);
#if SPEC_DEC
    static_assert(warpTile.y <= SharedMem::XSmemBuffer::rows);
#else
    static_assert(nbValidRows <= SharedMem::XSmemBuffer::rows);
#endif
    constexpr uint32_t nbIters = divUp(nbValidRows * SharedMem::XSmemBuffer::cols, warp_size);
#pragma unroll
    for (uint32_t i = 0; i < nbIters; i++)
    {
        uint32_t const flatIdx = warp_size * i + laneId();
        uint32_t const r = flatIdx / SharedMem::XSmemBuffer::cols;
        uint32_t const c = flatIdx % SharedMem::XSmemBuffer::cols;
        assert(r < SharedMem::XSmemBuffer::rows);
        LdGrain const data = src.template at<true>(r, c);

        uint32_t const m = dstOffset.y + r;
        uint32_t const n = exactDiv(dstOffset.x, grainBytes / inputElemSize) + c;
#if SPEC_DEC
        if (r >= nbValidHeadTokens)
        {
#else
        if (nbValidRows * SharedMem::XSmemBuffer::cols % warp_size != 0 && m >= nbValidRows)
        {
#endif
            break;
        }
        assert(m < nbValidRows);
#if SPEC_DEC
        uint32_t const idxBeam = 0;
        uint32_t const idxInGrp = m;
        uint32_t const tokenIdx = idxInGrp / headGrpSize;
        uint32_t const headIdx = idxInGrp % headGrpSize;
        assert(idxBeam < beamWidth);
        uint32_t const idxHead = idxHeadGrpOffset + tokenIdx * nbQHeads + headIdx;
        assert(idxHead < nbValidHeadTokens * nbQHeads);
#else
        uint32_t const idxBeam = m / headGrpSize;
        uint32_t const idxInGrp = m % headGrpSize;
        assert(idxBeam < beamWidth);
        uint32_t const idxHead = headGrpSize * idxHeadGrp + idxInGrp;
        assert(idxHead < nbQHeads);
#endif
        assert(n < paddedInputHeadBytes / grainBytes);
        if (!isHeadPadded || n < ioHeadBytes / grainBytes)
        {
            auto const outVec
                = convert<OutputHead::Elem>(reinterpret_cast<Vec<InputElem, inputElemsPerGrain> const&>(data));
            reinterpret_cast<Vec<mha::decay_t<decltype(outVec)>, exactDiv(ioHeadBytes, grainBytes)>&>(
                dst[nbQHeads * idxBeam + idxHead])[n]
                = outVec;
        }
    }
}

// MMA instruction expansion in GEMM k-dim and m/n-dim, with b16 8x8 as baseline
template <uint32_t kEx_, uint32_t mnEx_>
struct InstInMat
{
    static constexpr uint32_t kEx = kEx_;
    static constexpr uint32_t mnEx = mnEx_;
    uint32_t data[kEx][mnEx];
};

template <uint32_t kEx, uint32_t mnEx, bool transOuter>
using InstInMatWTrans = InstInMat<transOuter ? mnEx : kEx, transOuter ? kEx : mnEx>;

//@fixme: for B-mat, use InstInMat<2, 1>[2] instead.

// kEx is for srcCol and mnEx is for srcRow, before transpose.
// rowBeg/colBeg are in src indices
// note that grainBytes-byte swizzling per 128-byte or per row(>=128byte) is applied when loading to avoid bank
// conflict. transOuter: transpose InstInMat with 8x8 b16 matrices as elements unchanged. transInner: transpose the
// elements, i.e. the 8x8 b16 matrices. transOuter=true and transInner=false is for B matrix of 16816. It actually loads
// two 8x16 B matrices for two instructions. transOuter=false and transInner=false is for A matrix of 16816.
template <uint32_t kEx, uint32_t mnEx, bool transOuter, bool transInner, uint32_t srcRows, uint32_t srcCols>
__device__ inline InstInMatWTrans<kEx, mnEx, transOuter> loadInstInMat(
    Warp const& warp, Array2D<LdGrain, srcRows, srcCols> const& src, uint32_t rowOffset, uint32_t colOffset)
{
    static_assert(kEx * mnEx == 4, "implemented only for ldmatrix.x4 for now");
    using Dst = InstInMatWTrans<kEx, mnEx, transOuter>;
    assert(rowOffset % (8 * mnEx) == 0 && colOffset % kEx == 0);
    uint32_t const idx = laneId() / 8;
    uint32_t const idxKEx = idx / Dst::mnEx;
    uint32_t const idxMNEx = idx % Dst::mnEx;
    uint32_t const srcIdxKEx = (transOuter ? idxMNEx : idxKEx);
    uint32_t const srcIdxMNEx = (transOuter ? idxKEx : idxMNEx);

    LdGrain const* const ptr = &src.template at<true>(rowOffset + 8 * srcIdxMNEx + laneId() % 8, colOffset + srcIdxKEx);

    Vec<uint32_t, 4> const data = ldmatrix_4x<transInner>(warp, ptr);
    static_assert(sizeof(Dst) == sizeof(data));
    Dst dst;
#pragma unroll
    for (int i = 0; i < data.size; i++)
    {
        (&dst.data[0][0])[i] = data[i];
    }
    return dst;
}

template <typename T, uint32_t rows, uint32_t cols, bool transpose>
using Array2DWTrans = Array2D<T, transpose ? cols : rows, transpose ? rows : cols>;

// src rows/cols are in src indices
// dst rows/cols are in InstInMatWTrans
// row is contiguous and gemm-K dim.
// kEx combines with dstCols and mnEx combines with dstRows.
template <uint32_t kEx, uint32_t mnEx, uint32_t dstRows, uint32_t dstCols, bool transArr2D, bool transInstInMatOuter,
    bool transInstInMatInner, uint32_t srcRows, uint32_t srcCols /*in LdGrain*/>
__device__ inline Array2DWTrans<InstInMatWTrans<kEx, mnEx, transInstInMatOuter>, dstRows, dstCols, transArr2D>
loadMatrix(Warp const& warp, Array2D<LdGrain, srcRows, srcCols> const& src, uint32_t rowBeg, uint32_t colBeg)
{
    assert(rowBeg % (8 * mnEx * dstRows) == 0 && colBeg % (kEx * dstCols) == 0);
    Array2DWTrans<InstInMatWTrans<kEx, mnEx, transInstInMatOuter>, dstRows, dstCols, transArr2D> dst;
#pragma unroll
    for (uint32_t i = 0; i < dstRows; i++)
    {
#pragma unroll
        for (uint32_t j = 0; j < dstCols; j++)
        {
            (transArr2D ? dst(j, i) : dst(i, j)) = loadInstInMat<kEx, mnEx, transInstInMatOuter, transInstInMatInner>(
                warp, src, rowBeg + (mnEx * 8) * i, colBeg + kEx * j);
        }
    }
    return dst;
}

// acc is used as both input and output
// qColBeg is in the unit of LdGrain
// using KElemType = int8_t;
template <typename KElemType>
__device__ inline void smemQKPartGemm(
    Warp const& warp, WarpAcc& acc, SharedMem::QSmemBuffer const& q, uint32_t qColBeg, SharedMem::KSmemBuffer const& k)
{
    assert(qColBeg % (SharedMem::KSmemBuffer::cols) == 0);
    constexpr uint32_t kEx = 2;
    constexpr uint32_t mnEx = 2;
    static_assert(mha::is_same_v<InputElem, half> || mha::is_same_v<InputElem, __hip_bfloat16>, "not implemented");
    static_assert((mha::is_same_v<KElemType, half> || mha::is_same_v<KElemType, __hip_bfloat16>
                      || mha::is_same_v<KElemType, int8_t> || mha::is_same_v<KElemType, __hip_fp8_e4m3_fnuz>),
        "not implemented");
    constexpr uint32_t nbInstInMatPerSliceInGemmKDim = 1;
    constexpr uint32_t kElemSize = sizeof(KElemType);
    constexpr uint32_t elemsPerKHeadPart = exactDiv(kHeadPartBytes, kElemSize);
    constexpr uint32_t gemmKSplit = exactDiv(elemsPerKHeadPart, 8 * kEx * nbInstInMatPerSliceInGemmKDim);

    // @fixme: check if compiler mixes LDS+HMMA and does prefetch properly. We are not doing prefetch explicitly. But we
    // do fully unroll and expect compiler to do that for us.
    constexpr uint32_t nbUnroll = cacheElemSize == 2 ? gemmKSplit : 2;
#pragma unroll(nbUnroll)
    for (uint32_t s = 0; s < gemmKSplit; s++)
    {
        // load q
        constexpr uint32_t qSliceRows = exactDiv(warpTile.y, 8 * mnEx); // in InstInMat
        constexpr uint32_t qSliceCols = nbInstInMatPerSliceInGemmKDim;
        Array2D<InstInMat<kEx, mnEx>, qSliceRows, qSliceCols> const qSlice
            = loadMatrix<kEx, mnEx, qSliceRows, qSliceCols, false, false, false>(
                warp, q, 0, qColBeg + kEx * qSliceCols * s);
        // load k
        constexpr uint32_t cvtExp = exactDiv(inputElemSize, kElemSize);
        constexpr uint32_t mnExK = mnEx * cvtExp;
        constexpr uint32_t kExK = exactDiv(kEx, cvtExp);
        constexpr uint32_t kSliceRows = exactDiv(warpTile.x, 8 * mnExK); // in InstInMat
        constexpr uint32_t kSliceCols = nbInstInMatPerSliceInGemmKDim;
        Array2D<InstInMat<mnExK, kExK>, kSliceRows, kSliceCols> const kSliceOrig
            = loadMatrix<kExK, mnExK, kSliceRows, kSliceCols, false, true, false>(warp, k, 0, kExK * kSliceCols * s);
        auto const kSlice = [&]() -> Array2D<InstInMat<mnExK, kEx>, kSliceRows, kSliceCols>
        {
            if constexpr (mha::is_same_v<InputElem, KElemType>)
            {
                return kSliceOrig;
            }
            else if constexpr ((mha::is_same_v<KElemType, int8_t> || mha::is_same_v<KElemType, __hip_fp8_e4m3_fnuz>) )
            {
                Array2D<InstInMat<mnExK, kEx>, kSliceRows, kSliceCols> ret;
#pragma unroll
                for (uint32_t m = 0; m < kSliceRows; m++)
                {
#pragma unroll
                    for (uint32_t n = 0; n < kSliceCols; n++)
                    {
#pragma unroll
                        for (uint32_t i = 0; i < mnExK; i++)
                        {
#pragma unroll
                            for (uint32_t j = 0; j < kExK; j++)
                            {
                                auto const data
                                    = convertKCacheWordToF16<InputElem, KElemType>(kSliceOrig(m, n).data[i][j]);
                                ret(m, n).data[i][j * cvtExp] = data[0];
                                ret(m, n).data[i][j * cvtExp + 1] = data[1];
                            }
                        }
                    }
                }
                return ret;
            }
            else
            {
                assert(!"not implemented");
                trap();
            }
        }();
// compute
#pragma unroll
        for (uint32_t i = 0; i < qSliceRows; i++)
        {
#pragma unroll
            for (uint32_t j = 0; j < kSliceRows; j++)
            {
                InstInMat<kEx, mnEx> const matrixA = qSlice(i, 0);
                InstInMat<mnExK, kEx> const matrixB = kSlice(j, 0);
#pragma unroll
                for (uint32_t n = 0; n < mnExK; n++)
                {
                    uint32_t const b[2][1] = {matrixB.data[n][0], matrixB.data[n][1]};
                    mma<InputElem>(acc(i, j * mnExK + n).data, matrixA.data, b);
                }
            }
        }
    }
}

// acc is used as both input and output
// v needs transpose
template <typename VElemType>
__device__ inline void smemXVPartGemm(Warp const& warp, WarpAcc& acc, bool skipXRowRescale,
    UniformRescaleMask xRowNeedRescaleMask, ThrdRegRowMax xRowScales, SharedMem::XSmemBuffer const& x,
    uint32_t idxVTilePerXTile, SharedMem::VSmemBuffer const& vt, uint32_t idxNSplit)
{
    static_assert(mha::is_same_v<InputElem, half> || mha::is_same_v<InputElem, __hip_bfloat16>, "not implemented");
    static_assert((mha::is_same_v<VElemType, half> || mha::is_same_v<VElemType, __hip_bfloat16>
                      || mha::is_same_v<VElemType, int8_t> || mha::is_same_v<VElemType, __hip_fp8_e4m3_fnuz>),
        "not implemented");
    constexpr uint32_t kEx = 2;
    constexpr uint32_t mnEx = 2;
    constexpr uint32_t nbInstInMatPerSliceInGemmKDim = 1;
    static_assert(SharedMem::XSmemBuffer::rows == 8 * InstAcc::rows * WarpAcc::rows);
    static_assert(
        grpLoadV || sizeof(SharedMem::VSmemBuffer::Elem) / cacheElemSize * SharedMem::VSmemBuffer::cols == warpTile.x);
    static_assert(
        !grpLoadV || sizeof(SharedMem::VSmemBuffer::Elem) / cacheElemSize * SharedMem::VSmemBuffer::cols == headElems);
    if (grpLoadV)
    {
        assert(idxNSplit < gemm1WarpsPerGrp);
    }
    else
    {
        assert(idxNSplit == 0);
    }
    constexpr uint32_t gemmKSplit = exactDiv(SharedMem::VSmemBuffer::rows, 8 * kEx * nbInstInMatPerSliceInGemmKDim);

    Vec<InputElem2, QuadRegRowMax::size> xRowScalesQuad;
    if (!enableMicroFastPath || !skipXRowRescale)
    {
        assertWarpConverged();
#if INPUT_FP16
        Vec<InputElem2, ThrdRegRowMax::size> const xRowScalesF16 = __float2half2_rn(xRowScales);
#else
        Vec<InputElem2, ThrdRegRowMax::size> const xRowScalesF16 = __float2bfloat162_rn(xRowScales);
#endif
        static_assert(sizeof(xRowScalesF16) == sizeof(ThrdRegRowMax));
        reinterpret_cast<QuadRegRowMax&>(xRowScalesQuad)
            = replicateForQuad(warp, reinterpret_cast<ThrdRegRowMax const&>(xRowScalesF16));
    }

// @fixme: check if compiler mixes LDS+HMMA and does prefetch properly. We are not doing prefetch explicitly. But we do
// fully unroll and expect compiler to do that for us.
#pragma unroll
    for (uint32_t s = 0; s < gemmKSplit; s++)
    {
        // load x
        constexpr uint32_t xSliceRows = exactDiv(warpTile.y, 8 * mnEx); // in InstInMat
        constexpr uint32_t xSliceCols = nbInstInMatPerSliceInGemmKDim;
        uint32_t const colBeg = SharedMem::XSmemBuffer::cols / nbCacheVTilesPerXTile * idxVTilePerXTile
            + exactDiv(inputElemSize * 8 * kEx * nbInstInMatPerSliceInGemmKDim, grainBytes) * s;
        Array2D<InstInMat<kEx, mnEx>, xSliceRows, xSliceCols> xSlice
            = loadMatrix<kEx, mnEx, xSliceRows, xSliceCols, false, false, false>(warp, x, 0u, colBeg);
        if (!enableMicroFastPath || !skipXRowRescale)
        {
#pragma unroll
            for (uint32_t m = 0; m < xSliceRows; m++)
            {
#pragma unroll
                for (uint32_t i = 0; i < mnEx; i++)
                {
                    uint32_t const r = m * mnEx + i;
#pragma unroll
                    for (uint32_t n = 0; n < xSliceCols; n++)
                    {
#pragma unroll
                        for (uint32_t j = 0; j < kEx; j++)
                        {
                            InputElem2& elem = reinterpret_cast<InputElem2&>(xSlice(m, n).data[j][i]);
                            elem = skipXRowRescale ? elem : elem * xRowScalesQuad[r];
                        }
                    }
                }
            }
        }
        // load v slice. rows and cols here are before transpose
        constexpr uint32_t mnExV = mnEx * cvtExpansion;
        constexpr uint32_t vSliceCols = exactDiv(warpTile.x, 8 * mnExV); // in InstInMat
        constexpr uint32_t vSliceRows = nbInstInMatPerSliceInGemmKDim;
        uint32_t const rowBeg = 8 * kEx * nbInstInMatPerSliceInGemmKDim * s;
        Array2D<InstInMat<mnEx, kEx>, vSliceCols, vSliceRows> const vSliceOrig
            = loadMatrix<mnEx, kEx, vSliceRows, vSliceCols, true, false, true>(
                warp, vt, rowBeg, mnEx * vSliceCols * idxNSplit);
        Array2D<InstInMat<mnExV, kEx>, vSliceCols, vSliceRows> const vSlice = [&]()
        {
            if constexpr (mha::is_same_v<InputElem, VElemType>)
            {
                return vSliceOrig;
            }
            else if constexpr ((mha::is_same_v<VElemType, int8_t> || mha::is_same_v<VElemType, __hip_fp8_e4m3_fnuz>) )
            {
                Array2D<InstInMat<mnExV, kEx>, vSliceCols, vSliceRows> ret;
#pragma unroll
                for (uint32_t m = 0; m < ret.rows; m++)
                {
#pragma unroll
                    for (uint32_t n = 0; n < ret.cols; n++)
                    {
                        auto const& src = vSliceOrig(m, n);
                        auto& dst = ret(m, n);
#pragma unroll
                        for (uint32_t i = 0; i < mnEx; i++)
                        {
#pragma unroll
                            for (uint32_t j = 0; j < kEx; j++)
                            {
                                auto const data = convertVCacheWordToF16<InputElem, VElemType>(src.data[i][j]);
#pragma unroll
                                for (uint32_t e = 0; e < cvtExpansion; e++)
                                {
                                    dst.data[i * cvtExpansion + e][j] = data[e];
                                }
                            }
                        }
                    }
                }
                return ret;
            }
            else
            {
                assert(!"not implemented");
                trap();
            }
        }();
// compute
#pragma unroll
        for (uint32_t i = 0; i < xSliceRows; i++)
        {
#pragma unroll
            for (uint32_t j = 0; j < vSliceCols; j++)
            {
                auto const& vInMat = vSlice(j, 0);
#pragma unroll
                for (uint32_t n = 0; n < mnExV; n++)
                {
                    mma<InputElem>(acc(i, j * mnExV + n).data, xSlice(i, 0).data,
                        reinterpret_cast<uint32_t const(&)[2][1]>(vInMat.data[n]));
                }
            }
        }
    }
}

__device__ inline void pickAccRowsForBeamSearch(Warp const& warp, WarpAcc& dst, WarpAcc const& src, bool isCtxTile,
    uint32_t idxBeam, void (*func)(float& d, float s))
{
    uint32_t const idxQuad = laneId() / 4;
    constexpr uint32_t nbQuads = warp_size / 4;
#pragma unroll
    for (uint32_t m = 0; m < WarpAcc::rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
#pragma unroll
            for (uint32_t n = 0; n < WarpAcc::cols; n++)
            {
#pragma unroll
                for (uint32_t j = 0; j < InstAcc::cols; j++)
                {
                    uint32_t const idxRow = instM * m + nbQuads * i + idxQuad;
                    if (isCtxTile || (idxRow >= headGrpSize * idxBeam && idxRow < headGrpSize * idxBeam + headGrpSize))
                    {
                        func(dst(m, n)(i, j), src(m, n)(i, j));
                    }
                }
            }
        }
    }
}

__device__ inline void rescaleAcc(
    Warp const& warp, WarpAcc& acc, UniformRescaleMask const& rescaleMask, ThrdRegRowMax const& rowScales)
{
    static_assert(WarpAcc::rows * InstAcc::rows * 8 <= ThrdRegRowMax::size * warp_size);
// QuadRegRowMax const quadRowScales = replicateForQuad(warp, rowScales);
#pragma unroll
    for (uint32_t m = 0; m < WarpAcc::rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
            uint32_t const r = m * InstAcc::rows + i; // in 8-row unit.
            bool const skip = enableMicroFastPath && ((rescaleMask[r / 4] & (0xFFU << 8 * r)) == 0);
            if (skip)
            { // @fixme: do we need this?
                continue;
            }
            // float const scale = quadRowScales[r]; // @fixme: see if this is faster than the line below.
            float const scale = replicateValForQuad(warp, rowScales, r);
#pragma unroll
            for (uint32_t n = 0; n < WarpAcc::cols; n++)
            {
#pragma unroll
                for (uint32_t j = 0; j < InstAcc::cols; j++)
                {
                    acc(m, n)(i, j) *= scale;
                }
            }
        }
    }
}

__device__ inline void rescaleAcc(Warp const& warp, WarpAcc& acc, float scale)
{
#pragma unroll
    for (uint32_t m = 0; m < acc.rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
#pragma unroll
            for (uint32_t n = 0; n < acc.cols; n++)
            {
#pragma unroll
                for (uint32_t j = 0; j < InstAcc::cols; j++)
                {
                    acc(m, n)(i, j) *= scale;
                }
            }
        }
    }
}

template <bool useFp32Acc, uint32_t nbWarps, uint32_t nbTiles, uint32_t rows, uint32_t cols>
__device__ inline void smemFp16ArraySum(
    uint32_t idxWarp, Array2D<LdGrain, rows, cols>& dst, Array2D<LdGrain, rows, cols> const tiles[nbTiles])
{
    constexpr uint32_t nbThrds = warp_size * nbWarps;
    uint32_t const tid = warp_size * idxWarp + laneId();
    constexpr uint32_t nbGrains = SharedMem::XSmemBuffer::rows * SharedMem::XSmemBuffer::cols;
    constexpr uint32_t nbGrainsPerThrd = exactDiv(nbGrains, nbThrds);
    using AccType = mha::conditional_t<useFp32Acc, float2, InputElem2>;

#pragma unroll
    for (uint32_t i = 0; i < nbGrainsPerThrd; i++)
    {
        Vec<AccType, LdGrain::size> result;
        result.fill(AccType{0, 0});
        uint32_t const idx = nbThrds * i + tid;
#pragma unroll
        for (uint32_t j = 0; j < nbTiles; j++)
        {
            auto const data = reinterpret_cast<Vec<InputElem2, LdGrain::size> const(&)[nbGrains]>(tiles[j])[idx];
            if constexpr (useFp32Acc)
            {
#if INPUT_FP16
                result = addFloat2(result, __half22float2(data));
#else
                result = addFloat2(result, __bfloat1622float2(data));
#endif
            }
            else
            {
                result = __hadd2_rn(result, data);
            }
        }
        auto& dstGrain = reinterpret_cast<Vec<InputElem2, LdGrain::size>(&)[nbGrains]>(dst)[idx];
        if constexpr (useFp32Acc)
        {
#if INPUT_FP16
            dstGrain = __float22half2_rn(result);
#else
            dstGrain = __floats2bfloat162_rn(result);
#endif
        }
        else
        {
            dstGrain = result;
        }
    }
}

template <uint32_t nbBuffers>
__device__ inline ThrdRegRowMax mergeRowMax(
    Warp const& warp, TinyPtr<SMemWarpRowMax> const rowMaxBuffers, uint32_t nbSubSeqPerSeq)
{
    ThrdRegRowMax regBuffers[nbBuffers];
    auto load = [&](uint32_t n)
    {
        assert(n < nbSubSeqPerSeq);
        regBuffers[n % nbBuffers] = rowMaxBuffers[n].loadToReg<false>(warp);
    };
#pragma unroll
    for (uint32_t i = 0; i < nbBuffers; i++)
    {
        if (i >= nbSubSeqPerSeq)
        {
            break;
        }
        load(i);
    }
    ThrdRegRowMax mergedRowMax = regBuffers[0];
    for (uint32_t n = 0; n < divUp(nbSubSeqPerSeq, nbBuffers); n++)
    {
#pragma unroll
        for (uint32_t i = 0; i < nbBuffers; i++)
        {
            uint32_t const idx = nbBuffers * n + i;
            if (idx >= nbSubSeqPerSeq)
            {
                break;
            }
            mergedRowMax = fmaxf(mergedRowMax, regBuffers[i]);
            uint32_t const idxNext = idx + nbBuffers;
            if (idxNext < nbSubSeqPerSeq)
            {
                load(idxNext);
            }
        }
    }
    return mergedRowMax;
}

#ifdef NDEBUG
__device__ __forceinline__
#else
CUBIN_EXPORT __global__
#endif
    void
    kernel_mha_impl(
#if SPEC_DEC
        uint32_t const qSeqLen, uint32_t const nbKHeads, uint32_t const headGrpSize,
        SeqLenDataType const* __restrict__ qCuSeqLens, // [nbReq + 1]
#else
        uint32_t const nbKHeads,
#endif
#if SLIDING_WINDOW
        uint32_t slidingWinSize,
#endif
        float qScale,
        OutputHead* __restrict__ const output, // [nbReq][beamWidth][nbQHeads]
#if LOW_PREC_OUTPUT
        float const* rcpOutScale,
#endif
        // NOTE: the input is actually Q buffer when integrated to TRT-LLM.
        IOHead const* __restrict__ const q, // [nbReq][beamWidth][nbQHeads],
#if SPEC_DEC
        MaskType const* __restrict__ mask,  // [qSeqLen, divUp(qSeqLen, 32)].
#endif
#ifdef NDEBUG
        KVCacheList<usePagedKVCache> const& cacheList,
#if BEAM_WIDTH > 1
        BeamSearchParams const& beamSearchParams,
#endif
#else
        KVCacheList<usePagedKVCache> const cacheList,
#if BEAM_WIDTH > 1
        BeamSearchParams const beamSearchParams,
#endif
#endif
        uint32_t const batchSize,
        float const* __restrict__ kvCacheScale, // Device memory scalar. Same scale for K and V cache. Used only for
                                                // int8/fp8 KV cache.
        uint32_t* __restrict__ semaphores = nullptr, void* __restrict__ scratch = nullptr)
{
    assert(allowMultiBlockMode || gridDim.x == 1);
    bool const isMultiBlock = allowMultiBlockMode && (gridDim.x != 1);
    uint32_t const nbSubSeqPerSeq = allowMultiBlockMode ? gridDim.x : 1;
    uint32_t const idxSubSeqInSeq = allowMultiBlockMode ? blockIdx.x : 0;
    assert(!isMultiBlock || (semaphores != nullptr && scratch != nullptr));

    static_assert(inputSeqLen == 1);
    // gridDim: x - K/V sequence-dim split; y - number of K or V heads per token; z - number of requests
    assert(gridDim.z == batchSize && gridDim.y == nbKHeads);
    extern __shared__ char smemByteBuf[];
    SharedMem& smem = *reinterpret_cast<SharedMem*>(&smemByteBuf[0]);

    uint32_t const idxReq = blockIdx.z;
#if SPEC_DEC
    // Variable query sequence length support.
    bool const variableQSeqLen = qCuSeqLens != nullptr;
    uint32_t const actualQSeqLen = variableQSeqLen ? uint32_t(qCuSeqLens[idxReq + 1] - qCuSeqLens[idxReq]) : qSeqLen;
    // Same as idxReq * qSeqLen if all sequences all the same.
    // Take different beams as different requests/sequences currently.
    uint32_t const reqSeqOffset = variableQSeqLen ? uint32_t(qCuSeqLens[idxReq]) : (qSeqLen * idxReq);

    uint32_t const nbVHeads = nbKHeads;
    uint32_t const nbQHeads = nbKHeads * headGrpSize;
    uint32_t const nbQHeadTokens = nbQHeads * actualQSeqLen;
    uint32_t const nbQKVHeads = nbQHeads + nbKHeads + nbVHeads;

    uint32_t const nbTokenBlocksPerGrp = gridDim.y / nbKHeads;
    uint32_t const idxHeadGrp = blockIdx.y / nbTokenBlocksPerGrp; // inside one request
    uint32_t const idxHeadTokenInGrp = (blockIdx.y % nbTokenBlocksPerGrp) * warpTile.y;
    uint32_t const totalNbHeadTokensInGrp = actualQSeqLen * headGrpSize;
    uint32_t const nbValidHeadTokens = idxHeadTokenInGrp > totalNbHeadTokensInGrp
        ? 0u
        : mha::min(totalNbHeadTokensInGrp - idxHeadTokenInGrp, rowsPerBlock);
    // Shift the mask ptr by batch_idx.
    mask += reqSeqOffset * divUp(qSeqLen, 32u);
#else
    uint32_t const nbQHeads = nbKHeads * headGrpSize;

    uint32_t const idxHeadGrp = blockIdx.y; // inside one request
#endif

    auto const ctaThrdId
        = threadIdx.x + warp_size * ctaShapeInWarps.x * (threadIdx.y + ctaShapeInWarps.y * threadIdx.z);
    assert(blockDim.x == ctaShapeInWarps.x * warp_size && blockDim.y == ctaShapeInWarps.y
        && blockDim.z == ctaShapeInWarps.z);
    auto const warp = this_warp();
    uint3 const warpIdx = getWarpIdx(warp);                                      // @fixme: use BoundedVal
    assert(warpIdx.x < ctaShapeInWarps.x && warpIdx.y < ctaShapeInWarps.y && warpIdx.z < ctaShapeInWarps.z);
    uint32_t const flatWarpIdPerRow = warpIdx.z * ctaShapeInWarps.x + warpIdx.x; // per ctaShapeInWarps.y value

    // initialize shared memory
    static_assert(persistentQ && ctaShapeInWarps.y == 1);
    if (ctaThrdId < ctaShapeInWarps.y)
    {
        init(&smem.qBarrier[ctaThrdId], warp_size * ctaShapeInWarps.x); // be sure to use .noinc
    }
    constexpr uint32_t cacheVTileSeqStride = cacheVTileSeqLen * gemm1NbWarpGrps;
    constexpr uint32_t nbXTilesPerXIter
        = cacheVTileSeqStride < warpTile.x ? 1 : exactDiv(cacheVTileSeqStride, warpTile.x);
    constexpr uint32_t nbXItersPerCtaTile = exactDiv(ctaShapeInWarps.x, nbXTilesPerXIter);
    constexpr uint32_t nbVItersPerXIter = exactDiv(warpTile.x * nbXTilesPerXIter, cacheVTileSeqStride);
    constexpr uint32_t nbWarpGrpsPerXTile = mha::min(nbCacheVTilesPerXTile, gemm1NbWarpGrps);
    static_assert(warpTile.x >= cacheVTileSeqLen, "not implemented yet");
    static_assert(ctaSize >= uint32_t(sizeof(smem.xBarriers) / sizeof(CtaBarrierPair)));
    if (ctaThrdId < uint32_t(sizeof(smem.xBarriers) / sizeof(CtaBarrierPair)))
    {
        (&smem.xBarriers[0][0])[ctaThrdId].initialize(warp_size, warp_size * gemm1WarpsPerGrp * nbWarpGrpsPerXTile);
    }
#if CTA_ROW_MAX_BACKWARD_METHOD == 3
    static_assert(ctaSize >= sizeof(smem.ctaRowMaxBwdBarriers) / sizeof(SharedMem::Barrier));
    if (ctaThrdId < sizeof(smem.ctaRowMaxBwdBarriers) / sizeof(SharedMem::Barrier))
    {
        init(&smem.ctaRowMaxBwdBarriers[0][0] + ctaThrdId, warp_size);
    }
#endif
#if CTA_ROW_MAX_BACKWARD_METHOD != 0
    static_assert(ctaSize >= sizeof(smem.ctaRowMax) / sizeof(float));
    if (ctaThrdId < sizeof(smem.ctaRowMax) / sizeof(float))
    {
        reinterpret_cast<float*>(&smem.ctaRowMax[0])[ctaThrdId] = safeInitRowMax;
    }
#endif
#if GRP_LOAD_V
    static_assert(ctaSize >= gemm1NbWarpGrps * nbVBuffers);
    if (ctaThrdId < gemm1NbWarpGrps * nbVBuffers)
    {
        init(smem.vBarrier(0, 0) + ctaThrdId, warp_size * gemm1WarpsPerGrp);
    }
    if (ctaThrdId < gemm1NbWarpGrps)
    {
        init(smem.warpGrpBar(ctaThrdId), warp_size * gemm1WarpsPerGrp);
    }
#endif
    __syncthreads();

#if ENABLE_FDL
    preExit();
    acqBulk();
#endif

    constexpr bool qkSwizzle = true;
    // load whole Q heads into shared memory
#if SPEC_DEC
    if (warpIdx.z == 0)
    {
        // map from idxQHead to idxHead in q input.
        auto const localQHeadTokenIdxMap
            = [nbQHeads, headGrpSize, reqSeqOffset, idxReq, idxHeadTokenInGrp](uint32_t idxHeadTokenLocal) -> uint32_t
        {
            assert(idxHeadTokenLocal < warpTile.y); // may be larger than nbValidRows, then the output does not matter.
            if constexpr (beamWidth == 1)
            {
                idxHeadTokenLocal += idxHeadTokenInGrp;
                uint32_t const tokenIdx = (idxHeadTokenLocal / headGrpSize);
                uint32_t const headIdx = idxHeadTokenLocal % headGrpSize;
                return tokenIdx * nbQHeads + headIdx;
            }
        };
        static_assert(nbValidRows <= warpTile.y);
        auto const srcBase = q;
        uint32_t const idxHeadTokenBeg = nbQHeads * reqSeqOffset + (idxHeadGrp * headGrpSize);
        TinyPtr<IOHead const> const src{srcBase, idxHeadTokenBeg};

        bool const isFullTile = (nbValidHeadTokens == warpTile.y);
        static_assert(nbQBuffers == 1);
        if (isFullTile)
        {
            copyHeadsAsync<PaddedInputHead, warpTile.y, ctaShapeInWarps.x, qkSwizzle, true, warpTile.y>(
                warpIdx.x, smem.q[warpIdx.y][0], src, nbValidHeadTokens, localQHeadTokenIdxMap);
        }
        else
        {
            copyHeadsAsync<PaddedInputHead, warpTile.y, ctaShapeInWarps.x, qkSwizzle, false, warpTile.y>(
                warpIdx.x, smem.q[warpIdx.y][0], src, nbValidHeadTokens, localQHeadTokenIdxMap);
        }

        ldgsts::barArrive(smem.qBarrier[warpIdx.y], true);
    }
#else
    if (warpIdx.z == 0)
    {
        // map from idxQHead to idxHead in q input.
        auto const localQHeadIdxMap = [nbQHeads, idxReq, idxHeadGrp](uint32_t idxHeadLocal) -> uint32_t
        {
            assert(idxHeadLocal < warpTile.y); // may be larger than nbValidRows, then the output does not matter.
            if constexpr (beamWidth == 1)
            {
                return idxHeadLocal;
            }
            uint32_t const idxBeam = idxHeadLocal / headGrpSize;
            uint32_t const result = idxHeadLocal + idxBeam * (nbQHeads - headGrpSize);
            uint32_t const idxQHeadInGrp = idxHeadLocal % headGrpSize;
            uint32_t const ref = nbQHeads * idxBeam + idxQHeadInGrp;
            assert(result == ref);
            unused(ref);
            return result;
        };
        static_assert(nbValidRows <= warpTile.y);
        auto const srcBase = q;
        // NOTE: read from Q buffer directly.
        uint32_t const idxHeadBeg = nbQHeads * beamWidth * idxReq + headGrpSize * idxHeadGrp;
        TinyPtr<IOHead const> const src{srcBase, idxHeadBeg};

        constexpr bool isFullTile = (nbValidRows == warpTile.y);
        static_assert(nbQBuffers == 1);
        copyHeadsAsync<PaddedInputHead, warpTile.y, ctaShapeInWarps.x, qkSwizzle, isFullTile, warpTile.y>(
            warpIdx.x, smem.q[warpIdx.y][0], src, nbValidRows, localQHeadIdxMap);
        ldgsts::barArrive(smem.qBarrier[warpIdx.y], true);
    }
#endif

    uint32_t const cacheSeqLen = getCacheSeqLen<usePagedKVCache>(cacheList, idxReq);
    static_assert(!(allowSlidingWindow && useSpecDec), "Sliding window is not yet supported in spec-dec mode");
#if SLIDING_WINDOW
    bool const rtIsReallySliding = (cacheSeqLen > slidingWinSize);
    uint32_t const nbTotalSkipTokens = rtIsReallySliding ? cacheSeqLen - slidingWinSize : 0;
#else
    constexpr bool rtIsReallySliding = false;
    constexpr uint32_t nbTotalSkipTokens = 0;
#endif
    uint32_t const nbSkipLeadingTiles = nbTotalSkipTokens / ctaTile.x;
    uint32_t const tile0NbSkipTokens = nbTotalSkipTokens % ctaTile.x;
#if USE_PAGED_KV_CACHE
    uint32_t const nbPages = divUp(cacheSeqLen, tokensPerPage);
    constexpr uint32_t nbPagesPerCtaTile = exactDiv(ctaTile.x, tokensPerPage);
#endif

    uint32_t const nbSeqIters = useKVCache ? divUp(cacheSeqLen, ctaTile.x) : 0;
#if SPEC_DEC
    uint32_t const nbSeqItersWithoutMask = (cacheSeqLen - qSeqLen) / ctaTile.x;
#endif

    uint32_t const seqStrideIters = nbSubSeqPerSeq;
    constexpr bool isKVCacheQuantized = (cacheElemSize < 2);
    uint32_t const seqIterInit = nbSkipLeadingTiles + idxSubSeqInSeq;
#if BEAM_WIDTH > 1
    uint32_t const nbCtxCtaTiles = beamSearchParams.ctxLenList[idxReq * beamWidth] / ctaTile.x;
#endif
    auto isConvergedTile = [&](uint32_t seqIter)
    {
#if BEAM_WIDTH == 1
        return true;
#else
        return seqIter < nbCtxCtaTiles;
#endif
    };
    if (warpIdx.z == 0)
    {
        float const qkScale = qScale * (isKVCacheQuantized ? kvCacheScale[0] : 1.f)
            * rsqrtf(validElemsPerHead); // qkScale is applied onto Q*K.T before softmax.
        CircIdx<nbKBuffers> idxCurrSMemKBuf{nbKBuffers - 1};
        auto const getSMemKTile = [&](uint32_t idx) -> SharedMem::KSmemBuffer& { return smem.k[warpIdx.x][idx]; };
#if BEAM_WIDTH > 1
        auto loadCacheIndir = [&](uint32_t seqIter, uint32_t idxBeam) mutable
        {
            auto& dst = smem.gemm0CacheIndir[warpIdx.x];
            uint32_t const offset = ctaTile.x * seqIter + warpTile.x * warpIdx.x;
            loadIndicesForBeamSearchAsync<1, warpTile.x>(
                0, dst, beamSearchParams, idxReq, idxBeam, offset, cacheSeqLen);
        };
        loadCacheIndir(seqIterInit, 0U);
#endif
#if USE_PAGED_KV_CACHE
#if BEAM_WIDTH == 1
        KCachePageIndices pageIdx = KCachePageIndices::filled(kBAD_PAGE_INDEX);
#endif
        auto loadPages = [&](uint32_t idxPage) mutable
        {
#if BEAM_WIDTH == 1
            uint32_t const idxBeam = 0;
            pageIdx = getPage<KCachePageIndices::size>(cacheList, true, idxReq, idxBeam, idxPage, nbPages);
#else
            auto& dst = smem.kCachePages[warpIdx.x];
            loadPagesForBeamSearchAsync<1>(0U, dst, cacheList, true, idxReq, idxPage, nbPages);
#endif
        };
        uint32_t idxPageBeg = nbPagesPerCtaTile * seqIterInit + warpIdx.x * warpTile.x / tokensPerPage;
        loadPages(idxPageBeg);
#else
        constexpr uint32_t idxBeamBase = 0U;
        uint32_t const cacheKSeqBaseOffset
            = cacheList.capacity * (idxHeadGrp + nbKHeads * 2 * (idxBeamBase + beamWidth * idxReq));
#endif
        auto loadKTilePart = [&](uint32_t seqIter, uint32_t idxBeam, uint32_t idxPart) mutable
        {
            assert(idxBeam < beamWidth);
            assert(seqIter % nbSubSeqPerSeq == seqIterInit % nbSubSeqPerSeq);
            auto const idxNextSMemKBuf = idxCurrSMemKBuf.next();
            auto& dst = getSMemKTile(idxNextSMemKBuf);
            uint32_t const dstHeadOffset = 0;
            uint32_t const seqOffset = ctaTile.x * seqIter + warpTile.x * warpIdx.x;
#if USE_PAGED_KV_CACHE
            uint32_t const idxHeadBeg = tokensPerPage * idxHeadGrp + seqOffset % tokensPerPage;
#if BEAM_WIDTH == 1
            HeadPtr<GMemCacheHead const, tokensPerPage, nbPagesPerWarpTile> const src{
                cacheList.pool, pageIdx, nbKHeads, idxHeadBeg};
#else
            IndexedHeadPtr<GMemCacheHead const, tokensPerPage, nbPagesPerWarpTile> const src{
                /*indices=*/smem.gemm0CacheIndir[warpIdx.x].data,
                /*pool=*/cacheList.pool,
                /*pageIndices=*/smem.kCachePages[warpIdx.x].data,
                /*nbKHeads=*/nbKHeads,
                /*offset=*/idxHeadBeg};
#endif
#else
            uint32_t const idxHeadBeg = cacheKSeqBaseOffset + seqOffset;
#if BEAM_WIDTH == 1
            TinyPtr<GMemCacheHead const> const src{cacheList.data, idxHeadBeg};
#else
            IndexedHeadPtr<GMemCacheHead const, 0, 0> const src{/*indices=*/smem.gemm0CacheIndir[warpIdx.x].data,
                /*pointer=*/cacheList.data,
                /*offset=*/idxHeadBeg,
                /*beamStride=*/cacheList.capacity * nbKHeads * 2};
            // trap();
            // assert("not implemented");
#endif
#endif
            // if (threadIdx.x == dbgPrintTid) {
            //     printf("K: seqIter=%u, idxBeam=%u, idxPart=%u: pointers={%p, %p}, indices={", seqIter, idxBeam,
            //     idxPart, src.pointers[0], src.pointers[1]); uint32_t const nbHeadsAvail = mha::min((seqOffset <
            //     cacheSeqLen ? cacheSeqLen - seqOffset : 0U), warpTile.x); for (int i = 0; i < nbHeadsAvail; i++) {
            //         printf("%u, ", src.indices[i]);
            //     }
            //     printf("}\n");
            // }
            bool const isFullTile = (seqIter + 1 < nbSeqIters);
            if (isFullTile)
            {
                copyPartialHeadsAsync<PaddedCacheHead, warpTile.x, nbPartsPerCacheKHead, qkSwizzle, true>(
                    warp, dst, dstHeadOffset, src, idxPart);
            }
            else
            {
                uint32_t const nbHeadsAvail
                    = (seqOffset < cacheSeqLen ? cacheSeqLen - seqOffset
                                               : 0U); // may also be full but it can be handled correctly anyway
                copyPartialHeadsAsync<PaddedCacheHead, warpTile.x, nbPartsPerCacheKHead, qkSwizzle, false>(
                    warp, dst, dstHeadOffset, src, idxPart, nbHeadsAvail);
            }
#if BEAM_WIDTH > 1
            // to make sure all threads has finished usage of cache indir and pages
            __syncwarp();
#endif
            if (idxPart + 1 == nbPartsPerCacheKHead)
            {
#if USE_PAGED_KV_CACHE
                bool const isForNextSeqIter = isConvergedTile(seqIter) || idxBeam == beamWidth - 1;
                if (isForNextSeqIter)
                {
                    idxPageBeg += nbPagesPerCtaTile * nbSubSeqPerSeq;
                    loadPages(idxPageBeg);
                }
#endif
#if BEAM_WIDTH > 1
                uint32_t idxBeamNext, seqIterDelta;
                mha::tie(idxBeamNext, seqIterDelta) = isConvergedTile(seqIter)
                    ? mha::tuple<uint32_t, uint32_t>(0U, 1U)
                    : carryLE<beamWidth>(idxBeam + 1, 0); // optimize for context cache
                loadCacheIndir(seqIter + seqStrideIters * seqIterDelta, idxBeamNext);
#endif
            }
        };

#if BEAM_WIDTH > 1
        ldgsts::commitGroup();
        ldgsts::waitGroup<0>();
        __syncwarp();
#endif
        loadKTilePart(seqIterInit, 0, 0);
        ldgsts::commitGroup(); // @fixme: do prefetch for next iter tile if last part
        idxCurrSMemKBuf++;

        auto& xBar = smem.xBarriers[warpIdx.y][warpIdx.x];
        bool xBarConsumedParityNext = false;

        bool qBarParityNext = false;
        auto& qBar = smem.qBarrier[warpIdx.y];
        qBar.wait_parity(qBarParityNext);
        qBarParityNext = !qBarParityNext;
        constexpr bool reorderForKCache = (useKVCache && inputElemSize == 2 && cacheElemSize == 1);
        if constexpr (reorderForKCache)
        {
            reorder16bQHeadsToMatch8bKCache<ctaShapeInWarps.x, qkSwizzle, true>(warpIdx.x, smem.q[warpIdx.y][0]);
            unused(qBar.arrive());
            qBar.wait_parity(qBarParityNext);
            qBarParityNext = !qBarParityNext;
            assertWarpConverged();
        }
#if CTA_ROW_MAX_BACKWARD_METHOD == 2
        ThrdRegRowMax initRowMax;
        initRowMax.fill(safeInitRowMax);
#endif
        for (uint32_t seqIter = seqIterInit; seqIter < nbSeqIters; seqIter += seqStrideIters)
        {
#if SHORT_SEQ_OPT
            if (ctaTile.x * seqIter + warpTile.x * warpIdx.x >= cacheSeqLen)
            {
                break;
            }
#endif
            auto runGemm0 = [&](auto elemK, uint32_t idxBeam)
            {
                assert(idxBeam < (isConvergedTile(seqIter) ? 1U : beamWidth));
                using KElemType = mha::decay_t<decltype(elemK)>;
                constexpr uint32_t elemsPerKHeadPart = exactDiv(kHeadPartBytes, sizeof(KElemType));
                constexpr uint32_t nbPartsPerKHead = exactDiv(headElems, elemsPerKHeadPart);
                // the accumulator
                WarpAcc acc{};
                constexpr uint32_t nbUnroll = (cacheElemSize == 2 ? nbPartsPerKHead : 1);
#pragma unroll(nbUnroll)
                for (uint32_t p = 0; p < nbPartsPerKHead; p++)
                {
                    constexpr bool syncKTileEarly
                        = (beamWidth > 1); // alternative is to use double buffer for cacheIndir and pages
                    if constexpr (syncKTileEarly)
                    {
                        // synchronize gemm0CacheIndir for the next loadKTilePart. the last loaded K tile is also
                        // sync'ed at the same time.
                        ldgsts::waitGroup<0>();
                        __syncwarp();
                    }
                    // prefetch next part into shared memory
                    uint32_t idxPartNext, idxBeamNext, nNextBias;
                    mha::tie(idxPartNext, idxBeamNext, nNextBias) = isConvergedTile(seqIter)
                        ? carryLE<nbPartsPerKHead, 1U>(p + 1, idxBeam, 0U)
                        : carryLE<nbPartsPerKHead, beamWidth>(p + 1, idxBeam, 0U);

                    loadKTilePart(seqIter + seqStrideIters * nNextBias, idxBeamNext, idxPartNext);
                    ldgsts::commitGroup();
                    // @fixme: do L2 cache prefetch for next iter tile if last part

                    // q is already synchronized
                    if constexpr (!syncKTileEarly)
                    {
                        // synchronize k
                        ldgsts::waitGroup<1>();
                    }
                    SharedMem::QSmemBuffer const& smemQ = smem.q[warpIdx.y][0];
                    constexpr uint32_t qOffsetPerPart = exactDiv(elemsPerKHeadPart, inputElemsPerGrain);
                    uint32_t const smemQOffset = qOffsetPerPart * p;
                    SharedMem::KSmemBuffer const& smemKPart = getSMemKTile(idxCurrSMemKBuf);
                    // #ifndef NDEGBUG
                    //                     for (uint32_t i = 0; i < exactDiv(smemKPart.rows * smemKPart.cols,
                    //                     warp_size); i++) {
                    //                         uint32_t const idx = warp_size * i + laneId();
                    //                         uint32_t const r = idx / smemKPart.cols;
                    //                         uint32_t const c = idx % smemKPart.cols;

                    //                         assert(smemKPart(r, c) == );
                    //                     }
                    // #endif
                    // do computation.
                    smemQKPartGemm<KElemType>(warp, acc, smemQ, smemQOffset, smemKPart);
                    idxCurrSMemKBuf++;
                }
                return acc;
            };
            WarpAcc acc;
            //@fixme: alternative is to use separate inner loop, which results in larger but maybe faster code.
            for (uint32_t idxBeam = 0; idxBeam < (isConvergedTile(seqIter) ? 1U : beamWidth); idxBeam++)
            {
                WarpAcc tmp;
                if constexpr (mha::is_same_v<CacheElem, InputElem>)
                {
                    tmp = runGemm0(CacheElem{}, idxBeam);
                }
                else
                {
                    tmp = runGemm0(CacheElem{}, idxBeam);
                }
                pickAccRowsForBeamSearch(
                    warp, acc, tmp, isConvergedTile(seqIter), idxBeam, [](float& d, float s) { d = s; });
            }
            // apply qkScale
            rescaleAcc(warp, acc, qkScale);
#if CTA_ROW_MAX_BACKWARD_METHOD == 0
            QuadRegRowMax initRowMaxQuad;
            initRowMaxQuad.fill(safeInitRowMax);
#elif CTA_ROW_MAX_BACKWARD_METHOD == 1
            // load hint
            xBar.consumed.wait_parity(getAndFlip(xBarConsumedParityNext));
            QuadRegRowMax initRowMaxQuad = smem.ctaRowMax[warpIdx.y][warpIdx.x].loadToRegForQuad<false>(warp);
#elif CTA_ROW_MAX_BACKWARD_METHOD == 2
            QuadRegRowMax initRowMaxQuad = replicateForQuad(warp, initRowMax);
#elif CTA_ROW_MAX_BACKWARD_METHOD == 3
            // load hint
            smem.ctaRowMaxBwdBarriers[warpIdx.y][warpIdx.x].wait_parity(xBarConsumedParityNext);
            QuadRegRowMax initRowMaxQuad = smem.ctaRowMax[warpIdx.y][warpIdx.x].loadToRegForQuad<false>(warp);
#elif CTA_ROW_MAX_BACKWARD_METHOD == 4
            // load hint
            QuadRegRowMax initRowMaxQuad = smem.ctaRowMax[warpIdx.y].loadToRegForQuad<true>(warp);
#endif
            // masking
            uint32_t const warpTileTokenBeg = ctaTile.x * seqIter + warpTile.x * warpIdx.x;
#if SPEC_DEC
            if (seqIter >= nbSeqItersWithoutMask)
            {
                uint32_t const nbValidCols = (warpTileTokenBeg < cacheSeqLen ? cacheSeqLen - warpTileTokenBeg : 0U);
                applyMaskFromInput(
                    warp, acc, mask, idxHeadTokenInGrp, nbValidCols, qSeqLen, actualQSeqLen, headGrpSize);
            }
#else
            bool const isFirstIter = (seqIter == nbSkipLeadingTiles);
            bool const needMaskLeading = (rtIsReallySliding && isFirstIter);
            bool const isLastIter = (seqIter + 1 == nbSeqIters);
            bool const needMaskTrailing = isLastIter && cacheSeqLen % ctaTile.x != 0;
            if (needMaskLeading || needMaskTrailing)
            {
                uint32_t const validTokenBeg = (!needMaskLeading || nbTotalSkipTokens < warpTileTokenBeg)
                    ? 0
                    : nbTotalSkipTokens - warpTileTokenBeg;
                uint32_t const validTokenEnd = (warpTileTokenBeg < cacheSeqLen ? cacheSeqLen - warpTileTokenBeg : 0U);
                if (validTokenBeg > 0 || validTokenEnd < warpTile.x)
                {
                    applyMask(warp, acc, validTokenBeg, validTokenEnd);
                }
            }
#endif

            // find max and update acc into exp(acc-max).
            QuadRegRowMax const regRowMax = warpTileOnlineSoftmax(warp, initRowMaxQuad, acc);

            // store result and max to shared memory.
            GemmOutRegTile const fp16Acc = toFp16(acc);
            QuadRegRowMax const regRowSum = computeRowSum(warp, fp16Acc);
#if CTA_ROW_MAX_BACKWARD_METHOD != 1
            xBar.consumed.wait_parity(getAndFlip(xBarConsumedParityNext));
#if CTA_ROW_MAX_BACKWARD_METHOD == 2
            initRowMax = smem.ctaRowMax[warpIdx.y][warpIdx.x].loadToReg<false>(warp);
#endif
#endif
            storeOrderedGemmOutTile(warp, smem.x[warpIdx.y][warpIdx.x], fp16Acc);
            smem.warpRowMax[warpIdx.y][warpIdx.x].storeFromReg<false>(warp, regRowMax);
            smem.warpRowSum[warpIdx.y][warpIdx.x].storeFromReg<false>(warp, regRowSum);
            unused(xBar.produced.arrive());
        }
    }
    else
    {
        assert(warpIdx.z == 1);
#if CTA_ROW_MAX_BACKWARD_METHOD == 3
        unused(smem.ctaRowMaxBwdBarriers[warpIdx.y][warpIdx.x].arrive());
#endif
        uint32_t const warpIdxInGrp = gemm1WarpIdxInGrp(warpIdx.x); // @fixme: use BoundedVal
        uint32_t const warpGrpIdx = gemm1WarpGrpIdx(warpIdx.x);     // @fixme: use BoundedVal
        auto* const pWarpGrpBar = smem.warpGrpBar(warpGrpIdx);
        ParityOrNone<grpLoadV> warpGrpBarParityNext{};
#if BEAM_WIDTH > 1
        auto loadCacheIndir = [&](uint32_t seqIter, uint32_t xIter, uint32_t vIter, uint32_t idxBeam) mutable
        {
            uint32_t const seqOffset = ctaTile.x * seqIter + warpTile.x * nbXTilesPerXIter * xIter
                + cacheVTileSeqStride * vIter + cacheVTileSeqLen * warpGrpIdx;
            auto& dst = smem.gemm1CacheIndir[grpLoadV ? warpGrpIdx : warpIdx.x];
            loadIndicesForBeamSearchAsync<grpLoadV ? gemm1WarpsPerGrp : 1U, cacheVTileSeqLen>(
                grpLoadV ? warpIdxInGrp : 0U, dst, beamSearchParams, idxReq, idxBeam, seqOffset, cacheSeqLen);
        };
        loadCacheIndir(seqIterInit, 0, 0, 0);
#endif
        unused(smem.xBarriers[warpIdx.y][warpIdx.x].consumed.arrive(gemm1WarpsPerGrp * nbWarpGrpsPerXTile));
        CircIdx<nbVBuffers> idxCurrSMemVBuf{nbVBuffers - 1};
        auto const getSmemVTile = [&](uint32_t idx) -> SharedMem::VSmemBuffer&
        { return smem.v[warpGrpIdx][grpLoadV ? 0 : warpIdxInGrp][idx]; };
        auto const getSmemVBar = [&](uint32_t idx) -> SharedMem::Barrier* { return smem.vBarrier(warpGrpIdx, idx); };
#if USE_PAGED_KV_CACHE
#if BEAM_WIDTH == 1
        VCachePageIndices pageIdx = VCachePageIndices::filled(kBAD_PAGE_INDEX);
#endif
        auto loadPages = [&](uint32_t idxPageBeg) mutable
        {
#if BEAM_WIDTH == 1
            uint32_t const idxBeam = 0;
            pageIdx = getPage<VCachePageIndices::size>(cacheList, false, idxReq, idxBeam, idxPageBeg, nbPages);
#else
            auto& dst = smem.vCachePages[grpLoadV ? warpGrpIdx : warpIdx.x];
            loadPagesForBeamSearchAsync<grpLoadV ? gemm1WarpsPerGrp : 1U>(
                grpLoadV ? warpIdxInGrp : 0U, dst, cacheList, false, idxReq, idxPageBeg, nbPages);
#endif
        };
        uint32_t idxPageBeg = nbPagesPerCtaTile * seqIterInit + cacheVTileSeqLen * warpGrpIdx / tokensPerPage;
        loadPages(idxPageBeg);
#else
        uint32_t const idxBeamBase = 0;
        uint32_t const cacheVSeqBaseOffset
            = cacheList.capacity * (nbKHeads + idxHeadGrp + nbKHeads * 2 * (idxBeamBase + beamWidth * idxReq));
#endif
        auto nextStep = [&](uint32_t seqIter, uint32_t xIter, uint32_t vIter, uint32_t idxBeam)
        {
            uint32_t vIterNext, isNextBeam;
            mha::tie(vIterNext, isNextBeam) = carryLE<nbVItersPerXIter>(vIter + 1, 0);

            uint32_t idxBeamNext, xIterNext, nNextBias;
            mha::tie(idxBeamNext, xIterNext, nNextBias) = isConvergedTile(seqIter)
                ? carryLE<1, nbXItersPerCtaTile>(idxBeam + isNextBeam, xIter, 0)
                : carryLE<beamWidth, nbXItersPerCtaTile>(idxBeam + isNextBeam, xIter, 0);

            uint32_t const seqIterNext = seqIter + seqStrideIters * nNextBias;
            return mha::tuple<uint32_t, uint32_t, uint32_t, uint32_t>(seqIterNext, xIterNext, vIterNext, idxBeamNext);
        };
        auto loadVTilePart
            = [&](uint32_t seqIter, uint32_t xIter, uint32_t vIter,
                  uint32_t idxBeam) mutable { // @fixme: merge three iteration parameters into idxVTileGlb.
                  assert(idxBeam < beamWidth);
                  assert(seqIter % nbSubSeqPerSeq == seqIterInit % nbSubSeqPerSeq);
                  auto const idxNextSMemVBuf = idxCurrSMemVBuf.next();
                  auto& dst = getSmemVTile(idxNextSMemVBuf);
                  uint32_t const dstHeadOffset = 0;
                  constexpr bool vSwizzle = true;

                  uint32_t const seqOffset = ctaTile.x * seqIter + warpTile.x * nbXTilesPerXIter * xIter
                      + cacheVTileSeqStride * vIter + cacheVTileSeqLen * warpGrpIdx;
#if USE_PAGED_KV_CACHE
                  uint32_t const idxHeadBeg = tokensPerPage * idxHeadGrp + seqOffset % tokensPerPage;
#if BEAM_WIDTH == 1
                  HeadPtr<GMemCacheHead const, tokensPerPage, nbPagesPerVTile> const src{
                      cacheList.pool, pageIdx, nbKHeads, idxHeadBeg};
#else
                  IndexedHeadPtr<GMemCacheHead const, tokensPerPage, nbPagesPerVTile> const src{
                      /*indices=*/smem.gemm1CacheIndir[grpLoadV ? warpGrpIdx : warpIdx.x].data,
                      /*pool=*/cacheList.pool,
                      /*pageIndices=*/smem.vCachePages[grpLoadV ? warpGrpIdx : warpIdx.x].data,
                      /*nbKHeads=*/nbKHeads,
                      /*offset=*/idxHeadBeg};
#endif
#else
                  uint32_t const idxHeadBeg = cacheVSeqBaseOffset + seqOffset;
#if BEAM_WIDTH == 1
                  TinyPtr<GMemCacheHead const> const src{cacheList.data, idxHeadBeg};
#else
                  IndexedHeadPtr<GMemCacheHead const, 0, 0> const src{
                      /*indices=*/smem.gemm1CacheIndir[grpLoadV ? warpGrpIdx : warpIdx.x].data,
                      /*pointer=*/cacheList.data,
                      /*offset=*/idxHeadBeg,
                      /*beamStride=*/cacheList.capacity * nbKHeads * 2};
#endif
#endif
            // if (threadIdx.x == dbgPrintTid) {
            //     printf("V: seqIter=%u, xIter=%u, idxBeam=%u, vIter=%u: pointers={%p, %p}, indices={", seqIter, xIter,
            //     idxBeam, vIter, src.pointers[0], src.pointers[1]); uint32_t const nbHeadsAvail = mha::min((seqOffset
            //     < cacheSeqLen ? cacheSeqLen - seqOffset : 0U), cacheVTileSeqLen); for (int i = 0; i < nbHeadsAvail;
            //     i++) {
            //         printf("%u, ", src.indices[i]);
            //     }
            //     printf("}\n");
            // }

#if GRP_LOAD_V
                  uint32_t const nbHeadsAvail = (seqIter + 1 < nbSeqIters)
                      ? cacheVTileSeqLen
                      : (seqOffset < cacheSeqLen ? cacheSeqLen - seqOffset
                                                 : 0U); // may also be full but it can be handled correctly anyway
                  copyHeadsAsync<PaddedCacheHead, cacheVTileSeqLen, gemm1WarpsPerGrp, vSwizzle, false>(
                      warpIdxInGrp, dst, src, nbHeadsAvail);
#else
                  uint32_t const nbHeadsAvail
                      = (seqOffset < cacheSeqLen ? cacheSeqLen - seqOffset
                                                 : 0U); // may also be full but it can be handled correctly anyway
                  bool const isFullTile = (seqIter + 1 < nbSeqIters);
                  if (isFullTile)
                  {
                      copyPartialHeadsAsync<PaddedCacheHead, cacheVTileSeqLen, gemm1WarpsPerGrp, vSwizzle, true>(
                          warp, dst, dstHeadOffset, src, warpIdxInGrp);
                  }
                  else
                  {
                      uint32_t const nbHeadsAvail
                          = (seqOffset < cacheSeqLen ? cacheSeqLen - seqOffset
                                                     : 0U); // may also be full but it can be handled correctly anyway
                      copyPartialHeadsAsync<PaddedCacheHead, cacheVTileSeqLen, gemm1WarpsPerGrp, vSwizzle, false>(
                          warp, dst, dstHeadOffset, src, warpIdxInGrp, mha::min(nbHeadsAvail, cacheVTileSeqLen));
                  }
#endif

#if BEAM_WIDTH > 1
                  // to make sure all threads has finished usage of cache indir and pages
                  unused(arrive<grpLoadV>(pWarpGrpBar));
                  wait_parity<grpLoadV>(pWarpGrpBar, getAndFlip<grpLoadV>(warpGrpBarParityNext));
#endif
#if USE_PAGED_KV_CACHE
                  constexpr uint32_t xIterSeqStride = cacheVTileSeqStride * nbVItersPerXIter;
                  if constexpr (xIterSeqStride <= tokensPerPage)
                  {
                      uint32_t const nbXItersPerPage = exactDiv(tokensPerPage, xIterSeqStride);
                      assert(nbXItersPerPage <= nbXItersPerCtaTile);
                      if (xIter % nbXItersPerPage == nbXItersPerPage - 1 && vIter == nbVItersPerXIter - 1
                          && (idxBeam == beamWidth - 1 || isConvergedTile(seqIter)))
                      {
                          auto const step = 1; // cacheVTileSeqLen * gemm1NbWarpGrps / tokensPerPage;
                          idxPageBeg += (idxPageBeg % nbPagesPerCtaTile == nbPagesPerCtaTile - 1
                                  ? nbPagesPerCtaTile * (nbSubSeqPerSeq - 1) + step
                                  : step);
                          assert(beamWidth == 1
                              || cacheVTileSeqStride <= tokensPerPage
                                  && "todo: need to substrate from idxPageBeg for beam switching");
                          loadPages(idxPageBeg);
                      }
                  }
                  else
                  {
                      assert(nbVItersPerXIter == 1);
                      if ((idxBeam == beamWidth - 1 || isConvergedTile(seqIter)) && vIter == nbVItersPerXIter - 1)
                      {
                          auto const step = exactDiv(xIterSeqStride, tokensPerPage);
                          idxPageBeg += (idxPageBeg % nbPagesPerCtaTile + step >= nbPagesPerCtaTile
                                  ? nbPagesPerCtaTile * (nbSubSeqPerSeq - 1) + step
                                  : step);
                          loadPages(idxPageBeg);
                      }
                  }
#endif
#if BEAM_WIDTH > 1
                  uint32_t seqIterNext, xIterNext, vIterNext, idxBeamNext;
                  mha::tie(seqIterNext, xIterNext, vIterNext, idxBeamNext) = nextStep(seqIter, xIter, vIter, idxBeam);
                  loadCacheIndir(seqIterNext, xIterNext, vIterNext, idxBeamNext);
#endif
              };
        auto commitVTileLoad = [&](uint32_t idxVBar)
        {
#if GRP_LOAD_V
            auto& bar = *getSmemVBar(idxVBar);
            ldgsts::barArrive(bar, true);
#else
            ldgsts::commitGroup();
#endif
        };
        auto syncVTileLoad = [&](uint32_t idxVBar, ParityOrNone<grpLoadV> parity, bool alreadyComplete)
        {
#if GRP_LOAD_V
            if (alreadyComplete)
            {
                return;
            }
            SharedMem::Barrier& bar = *getSmemVBar(idxVBar);
            bar.wait_parity(parity);
#else
            assert(!alreadyComplete);
            ldgsts::waitGroup<nbVBuffers - 1>();
#endif
        };
        auto testVTileLoad = [&](uint32_t idxVBar, ParityOrNone<grpLoadV> parity)
        { return test_wait_parity<grpLoadV>(getSmemVBar(idxVBar), parity); };

#if BEAM_WIDTH > 1
        // synchronize first page/cacheIndir loading to shared memory
        ldgsts::commitGroup();
        ldgsts::waitGroup<0>();
        unused(arrive<grpLoadV>(pWarpGrpBar));
        wait_parity<grpLoadV>(pWarpGrpBar, getAndFlip<grpLoadV>(warpGrpBarParityNext));
#endif

        loadVTilePart(seqIterInit, 0, 0, 0);
        commitVTileLoad(idxCurrSMemVBuf.next());
        idxCurrSMemVBuf++;
        ParityOrNone<grpLoadV> vBarParity{};
        // @fixme: do prefetch for next iter tile if last part

        ThrdRegRowMax globalRowMax;
        globalRowMax.fill(safeInitRowMax);
        ThrdRegRowMax globalRowSum;
        globalRowSum.fill(0);
        // the accumulator
        WarpAcc acc{};
        if (grpLoadV)
        {
            unused(pWarpGrpBar->arrive());
        }
        bool xBarProducedParityNext = false;
        for (uint32_t seqIter = seqIterInit; seqIter < nbSeqIters; seqIter += seqStrideIters)
        {
#pragma unroll
            for (uint32_t xIter = 0; xIter < nbXItersPerCtaTile; xIter++)
            {
                uint32_t const idxXTile = xIter * nbXTilesPerXIter + warpGrpIdx / nbCacheVTilesPerXTile;
                assert(idxXTile < ctaShapeInWarps.x);
#if SHORT_SEQ_OPT
                if (ctaTile.x * seqIter + warpTile.x * idxXTile >= cacheSeqLen)
                {
                    break;
                }
#endif
                auto const& smemXTile = smem.x[warpIdx.y][idxXTile];
                auto& xBar = smem.xBarriers[warpIdx.y][idxXTile];
                ThrdRegRowMax xRowScales;
                UniformRescaleMask xRowNeedRescaleMask; // expect storage in UR
                bool skipXRowRescale;
                for (uint32_t idxBeam = 0; idxBeam < (isConvergedTile(seqIter) ? 1U : beamWidth); idxBeam++)
                {
#pragma unroll
                    for (uint32_t vIter = 0; vIter < nbVItersPerXIter; vIter++)
                    {
                        bool const vTestConsumed = test_wait_parity<grpLoadV>(pWarpGrpBar, warpGrpBarParityNext);
                        constexpr bool syncVTileEarly
                            = (beamWidth > 1); // alternative is to use double buffer for cacheIndir and pages
                        bool vTestProduced = syncVTileEarly && testVTileLoad(idxCurrSMemVBuf, vBarParity);
                        auto isLastVBuf = [&] { return (idxCurrSMemVBuf == idxCurrSMemVBuf.nbBuffers - 1); };
                        uint32_t const idxVTileInsideXIter = gemm1NbWarpGrps * vIter + warpGrpIdx;
                        uint32_t const idxVTile = idxVTileInsideXIter % nbCacheVTilesPerXTile; // inside XTile.
                        assert(idxVTile < nbCacheVTilesPerXTile);
                        uint32_t nNext, xIterNext, vIterNext, idxBeamNext;
                        mha::tie(nNext, xIterNext, vIterNext, idxBeamNext) = nextStep(seqIter, xIter, vIter, idxBeam);
                        if constexpr (syncVTileEarly)
                        {
                            // sync early to make sure that cacheIndir and pages has been loaded. The last loaded V tile
                            // is also sync'ed at the same time.
                            syncVTileLoad(idxCurrSMemVBuf, vBarParity, vTestProduced);
                            if (idxCurrSMemVBuf == idxCurrSMemVBuf.nbBuffers - 1)
                            {
                                flip<grpLoadV>(vBarParity);
                            }
                        }
                        if (!vTestConsumed)
                        {
                            wait_parity<grpLoadV>(pWarpGrpBar, warpGrpBarParityNext);
                        }
                        flip<grpLoadV>(warpGrpBarParityNext);
                        loadVTilePart(nNext, xIterNext, vIterNext, idxBeamNext);
                        commitVTileLoad(idxCurrSMemVBuf.next());
                        // @fixme: do L2 cache prefetch for next iter tile

                        if constexpr (!syncVTileEarly)
                        {
                            vTestProduced = testVTileLoad(idxCurrSMemVBuf, vBarParity);
                        }

                        if (idxBeam == 0 && vIter == 0)
                        {
                            xBar.produced.wait_parity(xBarProducedParityNext);
                            auto const& smemRowMax = smem.warpRowMax[warpIdx.y][idxXTile];
                            auto const& smemRowSum = smem.warpRowSum[warpIdx.y][idxXTile];
                            // update globalRowMax
                            ThrdRegRowMax xTileRowMax;
                            ThrdRegRowMax xTileRowSum;
                            UniformRescaleMask needRescaleMask;
#pragma unroll
                            for (uint32_t i = 0; i < ThrdRegRowMax::size; i++)
                            {
                                xTileRowMax[i] = smemRowMax[warp_size * i + laneId()];
                                xTileRowSum[i] = smemRowSum[warp_size * i + laneId()];
                                assert(__ballot_sync(~0U, laneId() == 0) == 1U);
                                assert(__ballot_sync(~0U, laneId() == 0) == 1U);
                                needRescaleMask[i] = __ballot_sync(~0U, xTileRowMax[i] != globalRowMax[i]);
                            }
                            bool const skipAllRescale = !any(needRescaleMask);
                            if (skipAllRescale)
                            {
                                skipXRowRescale = true;
#if CTA_ROW_MAX_BACKWARD_METHOD == 3
                                if (idxXTile == warpIdx.x)
                                {
                                    unused(smem.ctaRowMaxBwdBarriers[warpIdx.y][warpIdx.x].arrive());
                                }
#endif
                            }
                            else
                            {
                                ThrdRegRowMax const globalRowMaxOld = globalRowMax;
                                UniformRescaleMask accRowNeedRescaleMask;
#pragma unroll
                                for (uint32_t i = 0; i < ThrdRegRowMax::size; i++)
                                {
                                    accRowNeedRescaleMask[i] = __ballot_sync(~0U, xTileRowMax[i] > globalRowMaxOld[i]);
                                    xRowNeedRescaleMask[i] = (needRescaleMask[i] & ~accRowNeedRescaleMask[i]);
                                    assert(xRowNeedRescaleMask[i]
                                        == __ballot_sync(~0U, xTileRowMax[i] < globalRowMaxOld[i]));
                                    globalRowMax[i] = fmaxf(globalRowMaxOld[i], xTileRowMax[i]);
                                }
                                skipXRowRescale = !any(xRowNeedRescaleMask);

#if CTA_ROW_MAX_BACKWARD_METHOD == 1 || CTA_ROW_MAX_BACKWARD_METHOD == 2 || CTA_ROW_MAX_BACKWARD_METHOD == 3
                                // update smem.ctaRowMax.
                                if (idxXTile == warpIdx.x)
                                {
                                    smem.ctaRowMax[warpIdx.y][warpIdx.x].storeFromReg<false>(warp, globalRowMax);
#if CTA_ROW_MAX_BACKWARD_METHOD == 3
                                    unused(smem.ctaRowMaxBwdBarriers[warpIdx.y][warpIdx.x].arrive());
#endif
                                }
#elif CTA_ROW_MAX_BACKWARD_METHOD == 4
                                // update smem.ctaRowMax.
                                // smem.ctaRowMax[warpIdx.y].storeFromReg<true>(warp, globalRowMax);
                                smem.ctaRowMax[warpIdx.y].atomicMaxUpdate(warp, globalRowMax);
#endif
                                // update row sum and acc
                                if (!enableMicroFastPath || any(accRowNeedRescaleMask))
                                {
                                    ThrdRegRowMax const accRowScales = expf(globalRowMaxOld - globalRowMax);
                                    globalRowSum = globalRowSum * accRowScales;
                                    // @fixme: when tmpAcc is used, this can be delayed.
                                    rescaleAcc(warp, acc, accRowNeedRescaleMask, accRowScales);
                                }
                                if (!enableMicroFastPath || !skipXRowRescale)
                                {
                                    xRowScales = skipXRowRescale ? xRowScales : expf(xTileRowMax - globalRowMax);
                                    xTileRowSum = skipXRowRescale ? xTileRowSum : xTileRowSum * xRowScales;
                                }
                            }
                            globalRowSum = globalRowSum + xTileRowSum;
                        }
                        if constexpr (!syncVTileEarly)
                        {
                            syncVTileLoad(idxCurrSMemVBuf, vBarParity, vTestProduced);
                            if (idxCurrSMemVBuf == idxCurrSMemVBuf.nbBuffers - 1)
                            {
                                flip<grpLoadV>(vBarParity);
                            }
                        }
                        auto const& smemVTile = getSmemVTile(idxCurrSMemVBuf);
                        // do computation from shared memory X and V tiles
#if BEAM_WIDTH == 1
                        smemXVPartGemm<CacheElem>(warp, acc, skipXRowRescale, xRowNeedRescaleMask, xRowScales,
                            smemXTile, idxVTile, smemVTile, grpLoadV ? warpIdxInGrp : 0);
#else
                        WarpAcc tmpAcc{};
                        smemXVPartGemm<CacheElem>(warp, tmpAcc, skipXRowRescale, xRowNeedRescaleMask, xRowScales,
                            smemXTile, idxVTile, smemVTile, grpLoadV ? warpIdxInGrp : 0);
                        pickAccRowsForBeamSearch(
                            warp, acc, tmpAcc, isConvergedTile(seqIter), idxBeam, [](float& d, float s) { d += s; });
#endif
                        if (grpLoadV)
                        {
                            unused(pWarpGrpBar->arrive());
                        }
                        idxCurrSMemVBuf++;
                    }
                } // idxBeam
                xBar.consumed.arrive();
            }     // xIter
            flip(xBarProducedParityNext);
        }         // seqIter

        auto const fullRescaleMask = UniformRescaleMask::filled(~0U);

        constexpr bool needMergeGlobal = (gemm1NbWarpGrps > 1 && nbXTilesPerXIter > 1);
        if constexpr (needMergeGlobal)
        {
            assert(gemm1NbWarpGrps != 1);
            __syncthreads();
            smem.warpRowMax[warpIdx.y][warpIdx.x].template storeFromReg<false>(warp, globalRowMax);
            smem.warpRowSum[warpIdx.y][warpIdx.x].template storeFromReg<false>(warp, globalRowSum);
            __syncthreads();
            for (uint32_t i = 1; i < nbXTilesPerXIter; i++)
            { // i = 0 is for self and we can skip
                static_assert(nbXTilesPerXIter * nbWarpGrpsPerXTile == gemm1NbWarpGrps);
                uint32_t const otherWarpGrpIdx = (warpGrpIdx + nbWarpGrpsPerXTile * i) % gemm1NbWarpGrps;
                uint32_t const otherWarpIdx = warpIdxInGrp + gemm1WarpsPerGrp * otherWarpGrpIdx;
                assert(all(smem.warpRowMax[warpIdx.y][otherWarpIdx].template loadToReg<false>(warp)
                    == smem.warpRowMax[warpIdx.y][otherWarpIdx - warpIdxInGrp].template loadToReg<false>(warp)));
                auto const otherRowMax = smem.warpRowMax[warpIdx.y][otherWarpIdx].template loadToReg<false>(warp);
                auto const otherRowSum = smem.warpRowSum[warpIdx.y][otherWarpIdx].template loadToReg<false>(warp);
                auto const globalRowMaxNew = fmaxf(globalRowMax, otherRowMax);
                auto const scaleForThis = expf(globalRowMax - globalRowMaxNew);
                auto const scaleForOther = expf(otherRowMax - globalRowMaxNew);
                rescaleAcc(warp, acc, fullRescaleMask, scaleForThis);
                globalRowSum = globalRowSum * scaleForThis + otherRowSum * scaleForOther;
                globalRowMax = globalRowMaxNew;
            }
        }

        float voScale = (isKVCacheQuantized ? kvCacheScale[0] : 1.F);
        if (seqIterInit < nbSeqIters)
        { // otherwise rcpRowSum will be NAN.
            ThrdRegRowMax const rcpRowSum = __frcp_rn(globalRowSum);
#if LOW_PREC_OUTPUT
            voScale *= rcpOutScale[0];
#endif
            rescaleAcc(warp, acc, fullRescaleMask, rcpRowSum * ThrdRegRowMax::filled(voScale));
        }
        GemmOutRegTile const outTile = toFp16(acc);

        auto mergeAndSaveOutTile = [&](GemmOutRegTile const& tile, bool reorder)
        {
            if constexpr (gemm1NbWarpGrps == 1)
            {
                // swizzle in shared memory and write output global memory
                auto& outSwizzleBuffer = smem.x[warpIdx.y][warpIdx.x];
                __syncthreads();
                storeGemmOutTile(warp, outSwizzleBuffer, tile, reorder);
                __syncwarp();
                return &outSwizzleBuffer;
            }
            else
            {
                __syncthreads();
                // store to shared memory, then merge groups.
                using PostProcSMem = SharedMem::XSmemBuffer[ctaShapeInWarps.y][gemm1WarpsPerGrp][gemm1NbWarpGrps];
                static_assert(sizeof(PostProcSMem) <= smemSize);
                SharedMem::XSmemBuffer(&postSMem)[gemm1NbWarpGrps]
                    = reinterpret_cast<PostProcSMem&>(smem)[warpIdx.y][warpIdxInGrp];
                storeGemmOutTile(warp, postSMem[warpGrpIdx], tile, reorder);
                __syncthreads();
                smemFp16ArraySum<false, gemm1NbWarpGrps, gemm1NbWarpGrps>(warpGrpIdx, postSMem[0], postSMem);
                __syncthreads();
                return &postSMem[0];
            }
        };

        // merge results from different warp groups
        SharedMem::XSmemBuffer* smemOutTile = mergeAndSaveOutTile(outTile, inputElemSize == 2 && cacheElemSize == 1);
        if (isMultiBlock)
        {
            static_assert(ctaShapeInWarps.y == 1, "not implemented");
#if SPEC_DEC
            // Includes both kHeads and qTokens.
            uint32_t const nbIndepHeadTokens = gridDim.y;
            uint32_t const indepHeadTokenIdx = blockIdx.y;
            uint32_t const nbSeq = nbIndepHeadTokens * batchSize;
#else
            uint32_t const nbSeq = nbKHeads * batchSize;
#endif
            uint32_t const nbSubSeq = nbSubSeqPerSeq * nbSeq;
            MemSegmenter<false> segmenter{scratch};

#if SPEC_DEC
            uint32_t const idxSeq = nbIndepHeadTokens * idxReq + indepHeadTokenIdx;
#else
            uint32_t const idxSeq = nbKHeads * idxReq + idxHeadGrp;
#endif
            uint32_t const idxBufBase = nbSubSeqPerSeq * idxSeq;
            uint32_t const idxBuf = idxBufBase + idxSubSeqInSeq;
            // copy row max/sum
            TinyPtr<SMemWarpRowMax> const rowMaxBuffers = segmenter.newSeg<SMemWarpRowMax>(nbSubSeq);
            TinyPtr<SMemWarpRowMax> const rowSumBuffers = segmenter.newSeg<SMemWarpRowMax>(nbSubSeq);
            if (warpGrpIdx == 0 && warpIdxInGrp == 0)
            {
                rowMaxBuffers[idxBuf].storeFromReg<false>(warp, globalRowMax);
                rowSumBuffers[idxBuf].storeFromReg<false>(warp, globalRowSum);
            }
            using ScratchBuf = Array2D<LdGrain, nbValidRows, SharedMem::XSmemBuffer::cols>;
            TinyPtr<Vec<ScratchBuf, gemm1WarpsPerGrp>> const scratchBuffers
                = segmenter.newSeg<Vec<ScratchBuf, gemm1WarpsPerGrp>>(nbSubSeq);
            // copy output to scratch
            copyGrains<false, nbValidRows * ScratchBuf::cols, gemm1NbWarpGrps>(
                warpGrpIdx, &scratchBuffers[idxBuf][warpIdxInGrp](0, 0), &(*smemOutTile)(0, 0));
            __syncthreads();
            constexpr uint32_t nbTileBuffers = 2;

            struct MultiBlockSMem
            {
                bool isLastCta;

                struct MBBuf
                {
                    SMemWarpRowMax rowMax;
                    SMemWarpRowMax rowSum;
                    SharedMem::XSmemBuffer tiles[gemm1NbWarpGrps][gemm1WarpsPerGrp][nbTileBuffers];
                    SMemWarpRowMax tileRowMax[gemm1NbWarpGrps][gemm1WarpsPerGrp][nbTileBuffers];
                    SMemWarpRowMax tileRowSums[gemm1NbWarpGrps][gemm1WarpsPerGrp][nbTileBuffers];
                    SMemWarpRowMax mergedRowSum[gemm1NbWarpGrps];
                };

                MBBuf storage[ctaShapeInWarps.y];
            };

            static_assert(sizeof(MultiBlockSMem) <= smemSize);
            MultiBlockSMem& mbsmem = reinterpret_cast<MultiBlockSMem&>(smem);
            // increase the semaphore by 1
            if (warpIdx.y == 0 && warpGrpIdx == 0 && warpIdxInGrp == 0 && laneId() == 0)
            {
                uint32_t old;
                uint32_t const lastOld = nbSubSeqPerSeq - 1;
                asm volatile("atom.acq_rel.gpu.global.inc.u32 %0, [%1], %2;\n"
                             : "=r"(old)
                             : "l"(&semaphores[idxSeq]), "r"(lastOld));
                assert(old < nbSubSeqPerSeq);
                mbsmem.isLastCta = (old == lastOld);
            }
            __syncthreads();

            // merge if we are the last CTA.
            bool const isLastCta = mbsmem.isLastCta;
            if (isLastCta)
            {
                MultiBlockSMem::MBBuf& mbbuf = mbsmem.storage[warpIdx.y];
                SMemWarpRowMax& smemRowMax = reinterpret_cast<SMemWarpRowMax&>(smem);
                // get row max.
                if (warpIdx.x == 0)
                {
                    ThrdRegRowMax const mergedRowMax = mergeRowMax<8>(warp, rowMaxBuffers + idxBufBase, nbSubSeqPerSeq);
                    smemRowMax.storeFromReg<false>(warp, mergedRowMax);
                }
                __syncthreads();
                ThrdRegRowMax const mergedRowMax = smemRowMax.loadToReg<false>(warp);

                // rescale and accumulate
                auto getTileBuf = [&](auto& buffers, uint32_t d) -> decltype(buffers[0][0][0])&
                { return buffers[warpGrpIdx][warpIdxInGrp][d]; };
                auto loadBufAsync = [&](uint32_t n)
                {
                    uint32_t const d = n / gemm1NbWarpGrps % nbTileBuffers;
                    SharedMem::XSmemBuffer& dstTile = getTileBuf(mbbuf.tiles, d);
                    SMemWarpRowMax& dstRowSum = getTileBuf(mbbuf.tileRowSums, d);
                    SMemWarpRowMax& dstRowMax = getTileBuf(mbbuf.tileRowMax, d);
                    copyGrains<true, sizeof(ScratchBuf) / grainBytes, 1, true>(
                        0, &dstTile(0, 0), &scratchBuffers[idxBufBase + n][warpIdxInGrp](0, 0));
                    constexpr uint32_t nbGrainsPerRowMaxBuf = exactDiv(sizeof(SMemWarpRowMax), grainBytes);
                    copyGrains<true, roundUp(nbGrainsPerRowMaxBuf, 32u), 1, nbGrainsPerRowMaxBuf % 32 == 0>(0,
                        reinterpret_cast<LdGrain*>(&dstRowSum),
                        reinterpret_cast<LdGrain const*>(&rowSumBuffers[idxBufBase + n]), nbGrainsPerRowMaxBuf);
                    copyGrains<true, roundUp(nbGrainsPerRowMaxBuf, 32u), 1, nbGrainsPerRowMaxBuf % 32 == 0>(0,
                        reinterpret_cast<LdGrain*>(&dstRowMax),
                        reinterpret_cast<LdGrain const*>(&rowMaxBuffers[idxBufBase + n]), nbGrainsPerRowMaxBuf);
                };
                loadBufAsync(warpGrpIdx);
                ldgsts::commitGroup();
                WarpAcc sumAcc{};
                ThrdRegRowMax partialMergedRowSum{};
                for (uint32_t n = warpGrpIdx; n < nbSubSeqPerSeq; n += gemm1NbWarpGrps)
                {
                    if (n + gemm1NbWarpGrps < nbSubSeqPerSeq)
                    {
                        loadBufAsync(n + gemm1NbWarpGrps);
                    }
                    ldgsts::commitGroup();
                    ldgsts::waitGroup<1>();
                    uint32_t const d = n / gemm1NbWarpGrps % nbTileBuffers;
                    WarpAcc tile = toWarpAcc(loadGemmOutTile(warp, mbbuf.tiles[warpGrpIdx][warpIdxInGrp][d]));
                    ThrdRegRowMax const tileRowMax = getTileBuf(mbbuf.tileRowMax, d).loadToReg<false>(warp);
                    ThrdRegRowMax const tileRowSum = getTileBuf(mbbuf.tileRowSums, d).loadToReg<false>(warp);
                    ThrdRegRowMax const tileRowScales = expf(tileRowMax - mergedRowMax);
                    ThrdRegRowMax const scaledTileRowSum = tileRowSum * tileRowScales;
                    partialMergedRowSum = partialMergedRowSum + scaledTileRowSum;
                    assert(std::isfinite(partialMergedRowSum[0]));
                    rescaleAcc(warp, tile, fullRescaleMask, scaledTileRowSum);
                    sumAcc = sumAcc + tile;
                }

                ThrdRegRowMax mergedRowSum{};
                if (gemm1NbWarpGrps == 1)
                {
                    mergedRowSum = partialMergedRowSum;
                }
                else
                {
                    if (warpIdxInGrp == 0)
                    {
                        mbbuf.mergedRowSum[warpGrpIdx].storeFromReg<false>(warp, partialMergedRowSum);
                    }
                    __syncthreads();
#ifndef NDEBUG
                    assert((mbbuf.mergedRowSum[warpGrpIdx].loadToReg<false>(warp) == partialMergedRowSum)[0]);
                    __syncthreads();
#endif
#pragma unroll
                    for (uint32_t i = 0; i < gemm1NbWarpGrps; i++)
                    {
                        mergedRowSum = mergedRowSum + mbbuf.mergedRowSum[i].loadToReg<false>(warp);
                        assert(std::isfinite(mergedRowSum[0]));
                    }
                }
                __syncthreads();
                rescaleAcc(warp, sumAcc, fullRescaleMask, __frcp_rn(mergedRowSum));
                GemmOutRegTile const mergedOutTile = toFp16(sumAcc);
                smemOutTile = mergeAndSaveOutTile(mergedOutTile, false);
            }
        }
        if (warpGrpIdx == 0)
        {
#if SPEC_DEC
            copyOutputToGlobalMem(warp, &output[reqSeqOffset * nbQHeads], nbQHeads, headGrpSize,
                (idxHeadGrp * headGrpSize), nbValidHeadTokens,
                uint2{warpTile.x * warpIdxInGrp, nbValidRows * warpIdx.y + idxHeadTokenInGrp}, *smemOutTile);
#else
            copyOutputToGlobalMem(warp, &output[nbQHeads * beamWidth * idxReq], nbQHeads, idxHeadGrp,
                uint2{warpTile.x * warpIdxInGrp, nbValidRows * warpIdx.y}, *smemOutTile);
#endif
        }
    }
}

#if SPEC_DEC
#if __CUDA_ARCH__ == 900 && M_TILESIZE == 16
constexpr uint32_t nbCtaPerSM = 2;
#else
constexpr uint32_t nbCtaPerSM = 1;
#endif
#else
#if __CUDA_ARCH__ == 900
constexpr uint32_t nbCtaPerSM = 2;
#else
constexpr uint32_t nbCtaPerSM = 1;
#endif
#endif

CUBIN_EXPORT __device__ constexpr XQAKernelType kernelType = XQAKernelType::kAMPERE_WARP_SPECIALIZED;

#ifdef NDEBUG
CUBIN_EXPORT __global__ __launch_bounds__(256, nbCtaPerSM) void kernel_mha(
#if SPEC_DEC
    uint32_t const qSeqLen, uint32_t const nbKHeads, uint32_t const headGrpSize, SeqLenDataType const* qCuSeqLens,
#else
    uint32_t const nbKHeads,
#endif
#if SLIDING_WINDOW
    uint32_t slidingWinSize,
#endif
    float qScale,
    OutputHead* __restrict__ const output, // [nbReq][beamWidth][nbQHeads]
#if LOW_PREC_OUTPUT
    float const* rcpOutScale,
#endif
    IOHead const* __restrict__ const q, // [nbReq][beamWidth][nbQHeads],
#if SPEC_DEC
    MaskType const* __restrict__ mask,  // [qSeqLen, divUp(qSeqLen, 32))] uint2 (each bit represents mask for one col
                                        // position).
#endif
    KVCacheList<usePagedKVCache> const cacheList,
#if BEAM_WIDTH > 1
    BeamSearchParams const beamSearchParams,
#endif
    uint32_t const batchSize,
    float const* __restrict__ kvCacheScale, // Device memory scalar. Same scale for K and V cache. Used only for
                                            // int8/fp8 KV cache.
    uint32_t* __restrict__ semaphores = nullptr, void* __restrict__ scratch = nullptr)
{
#if SPEC_DEC
    kernel_mha_impl(qSeqLen, nbKHeads, headGrpSize, qCuSeqLens,
#else
    kernel_mha_impl(nbKHeads,
#endif
#if SLIDING_WINDOW
        slidingWinSize,
#endif
        qScale, output,
#if LOW_PREC_OUTPUT
        rcpOutScale,
#endif
        q,
#if SPEC_DEC
        mask,
#endif
        cacheList,
#if BEAM_WIDTH > 1
        beamSearchParams,
#endif
        batchSize, kvCacheScale, semaphores, scratch);
}
#else
static constexpr auto kernel_mha = kernel_mha_impl;
#endif

#ifndef GENERATE_CUBIN
void launchMHA(hipDeviceProp_t const& prop, uint32_t nbKHeads,
#if SLIDING_WINDOW
    uint32_t slidingWinSize,
#endif
    float qScale, OutputHead* output,
#if LOW_PREC_OUTPUT
    float const* rcpOutScale,
#endif
#if USE_INPUT_KV
    InputHead const* qkv,
#if ROPE_STYLE != 0
    Vec<float, validElemsPerHead> const* ropeCosSin,
#endif
#else
    InputHead const* q,
#endif
#if USE_PAGED_KV_CACHE
    GMemCacheHead* pool, // global pool of pages
    KVCachePageIndex const*
        kvCachePageList, // device pointer. shape: KVCachePageIndex[batchSize][beamWidth][2][maxNbPagesPerSeq].
#else
    GMemKVCacheHead* kvCacheData,
#endif
    uint32_t maxSeqLen, uint32_t const* seqLen,
#if BEAM_WIDTH > 1
    BeamSearchParams const& beamSearchParams,
#endif
    uint32_t batchSize,
    float const* __restrict__ kvCacheScale, // Device memory scalar. Same scale for K and V cache. Used only for
                                            // int8/fp8 KV cache.
#if SPEC_DEC
    SpecDecParams const& specDecParams,
#endif
    uint32_t* semaphores, void* scratch, hipStream_t stream)
{
#if SPEC_DEC
    auto const qSeqLen = specDecParams.qSeqLen;
    auto const qCuSeqLens = specDecParams.qCuSeqLens;
    auto const mask = specDecParams.mask;
#endif
#if USE_INPUT_KV
    throw std::runtime_error("not implemented");
#else
    static uint32_t const hostSmemSize = [&]()
    {
        uint32_t size;
        checkCuda(hipMemcpyFromSymbol(&size, HIP_SYMBOL(smemSize), sizeof(smemSize)));
        checkCuda(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_mha), hipFuncAttributeMaxDynamicSharedMemorySize, size));
        return size;
    }();
    uint32_t const nbVHeads = nbKHeads;
    uint32_t const nbQHeads = nbKHeads * headGrpSize;

    // const uint32_t nbSubSeqPerSeq = allowMultiBlockMode ? DBG_NB_CTAS_PER_SEQ : 1;
    uint32_t const nbSubSeqPerSeq = [&]() -> uint32_t
    {
        if (!allowMultiBlockMode)
        {
            return 1;
        }
        auto const env = std::getenv("XQA_NB_SUB_SEQ");
        if (env != nullptr)
        {
            int32_t const val = std::stoi(env);
            if (val > 0)
            {
                return val;
            }
        }
        return std::min<uint32_t>(
            std::max<uint32_t>(1U, prop.multiProcessorCount / (batchSize * nbKHeads)), divUp(maxSeqLen, ctaTile.x));
    }();
    // gridDim.z == batchSize && gridDim.y == nbKHeads && gridDim.x == nbSubSeqPerSeq
#if SPEC_DEC
    const uint32_t nbTokenBlocksPerGrp = divUp(qSeqLen * headGrpSize, rowsPerBlock);
    dim3 const dimGrid{nbSubSeqPerSeq, nbKHeads * nbTokenBlocksPerGrp, batchSize};
#else
    dim3 const dimGrid{nbSubSeqPerSeq, nbKHeads, batchSize};
#endif
    dim3 const dimCta{warp_size * ctaShapeInWarps.x, ctaShapeInWarps.y, ctaShapeInWarps.z};
    auto const launchCfg = makeLaunchConfig(dimGrid, dimCta, hostSmemSize, stream, ENABLE_FDL != 0);
#if USE_PAGED_KV_CACHE
    uint32_t const maxNbPagesPerSeq = exactDiv(maxSeqLen, tokensPerPage);
    KVCacheList<true> const cacheList{pool, kvCachePageList, seqLen, maxNbPagesPerSeq};
    cudaLaunchKernelEx(&launchCfg, kernel_mha,
#if SPEC_DEC
        qSeqLen, nbKHeads, headGrpSize, qCuSeqLens,
#else
        nbKHeads,
#endif
#if SLIDING_WINDOW
        slidingWinSize,
#endif
        qScale, output,
#if LOW_PREC_OUTPUT
        rcpOutScale,
#endif
        q,
#if SPEC_DEC
        mask,
#endif
        cacheList,
#if BEAM_WIDTH > 1
        beamSearchParams,
#endif
        batchSize, kvCacheScale, semaphores, scratch);
#else
    KVCacheList<false> const cacheList{kvCacheData, seqLen, maxSeqLen};
#ifndef NDEBUG
    kernel_mha<<<dimGrid, dimCta, hostSmemSize, stream>>>(
#else
    cudaLaunchKernelEx(&launchCfg, &kernel_mha,
#endif
#if SPEC_DEC
        qSeqLen, nbKHeads, headGrpSize, qCuSeqLens,
#else
        nbKHeads,
#endif
#if SLIDING_WINDOW
        slidingWinSize,
#endif
        qScale, output,
#if LOW_PREC_OUTPUT
        rcpOutScale,
#endif
        q,
#if SPEC_DEC
        mask,
#endif
        cacheList,
#if BEAM_WIDTH > 1
        beamSearchParams,
#endif
        batchSize, kvCacheScale, semaphores, scratch);
#endif
    checkCuda(hipPeekAtLastError());
#endif // USE_INPUT_KV
}
#endif
