#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: NVIDIA TensorRT Source Code License Agreement
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "defines.h"
#include "mha.h"
#if IS_MLA
#include "barriers.cuh"
#include "mhaUtils.cuh"
#include "mha_components.cuh"
#include "mha_stdheaders.cuh"
#include "mla_sm120.cuh"
#include "mma.cuh"
#include "tma.h"
#include "utils.cuh"
#include "utils.h"

#ifndef GENERATE_CUBIN
#include "hostUtils.h"
#include "tensorMap.h"
#include <hip/hip_runtime.h>
#endif

#define USE_REG_Q 1

__constant__ constexpr XQAKernelType kernelType = XQAKernelType::kSM120_MLA;

inline constexpr bool allowMultipleInputTokens = true;

inline constexpr uint32_t partElemsK = 64; // @fixme: change this to 128 to save L2 traffic
inline constexpr uint32_t nbKParts = exactDiv(validElemsPerKHead, partElemsK);
inline constexpr uint32_t nbQParts = nbKParts;

inline constexpr uint32_t tokensPerTile = 64;
inline constexpr uint32_t partElemsV = 128;
inline constexpr uint32_t nbVSplit = 2;
inline constexpr uint32_t gemm1V = exactDiv(validElemsPerVHead, nbVSplit);
inline constexpr uint32_t nbProducerCtasPerCga = nbVSplit;

inline constexpr uint32_t multiBlockMinNbTilesPerCta = 2;
inline constexpr uint32_t multiBlockMinNbTiles = multiBlockMinNbTilesPerCta * 2;

using MathElem = CacheElem;
inline constexpr uint32_t mathElemBytes = sizeof(MathElem);
inline constexpr uint32_t grainsPerPartK = exactDiv(partElemsK * mathElemBytes, grainBytes);

inline constexpr uint32_t grainElems = exactDiv(grainBytes, mathElemBytes);

inline constexpr float xScale = 1.f / kE4M3_MAX;
__constant__ constexpr float rcpXScale = kE4M3_MAX;

inline constexpr uint32_t nbRegsForIOWarps = 32;
inline constexpr uint32_t nbRegsForMathWarps = 232;

inline constexpr bool computeRowSumFromF8 = true;

struct KVTilePartLoader
{
#if USE_PAGED_KV_CACHE
    static_assert(tokensPerPage % tokensPerTile == 0 || tokensPerTile % tokensPerPage == 0);
    static inline constexpr uint32_t nbPagesPerTile
        = tokensPerTile >= tokensPerPage ? exactDiv(tokensPerTile, tokensPerPage) : 1;
#endif

    static inline constexpr uint32_t const nbKHeads = 1;
    KVCacheList<usePagedKVCache> const& cacheList;
    uint32_t const idxReq;
    static inline constexpr uint32_t const idxHeadGrp = 0;

    CUtensorMap const& tensorMap;
    // if greater than 1, then we need unrolling for the loading loop. Seems 1 is fine for latency.
    static inline constexpr uint32_t nbPageBuffers = 1;
#if USE_PAGED_KV_CACHE
    uint32_t const nbPages; // for bound check
    Vec<KVCachePageIndex, nbPagesPerTile> pageBuffers[nbPageBuffers];
    uint32_t idxTileRef;    // idxTile used to load the pages
#endif
    uint32_t const baseOffset;

    __device__ KVTilePartLoader(
        KVCacheList<usePagedKVCache> const& cacheList, uint32_t idxReq, CUtensorMap const& tensorMap
#if USE_PAGED_KV_CACHE
        ,
        uint32_t nbPages
#endif
    );
    // tensorMap is for one whole page ([nbKHeads*tokensPerPage][headElems]) or whole cache
    template <uint32_t nbTokens, uint32_t grainsPerPart, bool alignedForSwizzle>
    __device__ void loadData(Array2D<LdGrain, nbTokens, grainsPerPart, alignedForSwizzle>& dst, uint32_t idxTile,
        uint32_t idxElemBeg, CtaBarrier& bar, uint32_t idxPageBuf);

    __device__ void loadPages(uint32_t idxTile, uint32_t idxPageBuf);
};

__device__ inline KVTilePartLoader::KVTilePartLoader(
    KVCacheList<usePagedKVCache> const& cacheList, uint32_t idxReq, CUtensorMap const& tensorMap
#if USE_PAGED_KV_CACHE
    ,
    uint32_t nbPages
#endif
    )
    : cacheList{cacheList}
    , idxReq{idxReq}
    , tensorMap{tensorMap}
#if USE_PAGED_KV_CACHE
    , nbPages{nbPages}
    , baseOffset{((idxReq * beamWidth) * 2) * cacheList.maxNbPagesPerSeq}
#else
    , baseOffset{(idxReq * beamWidth) * 2}
#endif
{
}

// tensorMap is for one whole page ([nbKHeads*tokensPerPage][headElems]) or whole cache
template <uint32_t nbTokens, uint32_t grainsPerPart, bool alignedForSwizzle>
__device__ inline void KVTilePartLoader::loadData(Array2D<LdGrain, nbTokens, grainsPerPart, alignedForSwizzle>& dst,
    uint32_t idxTile, uint32_t idxElemBeg, CtaBarrier& bar, uint32_t idxPageBuf)
{
    static_assert(nbTokens == tokensPerTile);
#if USE_PAGED_KV_CACHE
    assert(idxTile == idxTileRef);
    auto const& pages = pageBuffers[idxPageBuf];
    if constexpr (nbTokens < tokensPerPage)
    {
        assert(nbPagesPerTile == 1);
        uint32_t const offset = nbTokens * (idxTile % exactDiv(tokensPerPage, nbTokens));
        if (warpElectSync())
        {
            tma::loadAsync(&dst, tensorMap, DimsLE<4>{idxElemBeg, offset, idxHeadGrp, (uint32_t) pages[0]}, bar);
        }
    }
    else
    {
#pragma unroll
        for (uint32_t i = 0; i < nbPagesPerTile; i++)
        {
            if (warpElectSync())
            {
                tma::loadAsync(&dst(tokensPerPage * i, 0), tensorMap,
                    DimsLE<4>{idxElemBeg, 0, idxHeadGrp, (uint32_t) pages[i]}, bar);
            }
        }
    }
#else
    if (warpElectSync())
    {
        tma::loadAsync(&dst, tensorMap, DimsLE<4>{idxElemBeg, nbTokens * idxTile, idxHeadGrp, baseOffset}, bar);
    }
#endif
}

__device__ inline void KVTilePartLoader::loadPages(uint32_t idxTile, uint32_t idxPageBuf)
{
#if USE_PAGED_KV_CACHE
    uint32_t const idxPageBeg
        = tokensPerTile >= tokensPerPage ? nbPagesPerTile * idxTile : idxTile / exactDiv(tokensPerPage, tokensPerTile);
    auto& pages = pageBuffers[idxPageBuf];
#pragma unroll
    for (uint32_t i = 0; i < nbPagesPerTile; i++)
    {
        uint32_t const idxPage = idxPageBeg + i;
        pages[i] = idxPage < nbPages ? cacheList.kvCachePageList[baseOffset + idxPage] : kBAD_PAGE_INDEX;
    }
    idxTileRef = idxTile;
#endif
}

using Mat16x32 = Vec<uint32_t, 4>;

template <uint32_t srcRows, uint32_t srcCols>
class Mat16x32Loader
{
public:
    using Src = Array2D<LdGrain, srcRows, srcCols>;

    // default r and c are for mat A.
    __device__ inline Mat16x32Loader(
        Src const& src, uint32_t baseRow, uint32_t idxInstK, uint32_t r = laneId() % 16, uint32_t c = laneId() / 16)
        : src{src}
        , baseRow{baseRow}
        , idxInstK{idxInstK}
        , r{r}
        , c{c}
        , basePtr{getPtrRef(0)}
    {
        static_assert((grainBytes * srcCols * qmmaShape.m) % 1024 == 0);
    }

    __device__ inline Mat16x32 load(uint32_t idxInstM) const
    {
        return ldmatrix<false, 4>(getPtr(idxInstM));
    }

    template <uint32_t tileM>
    __device__ inline Vec<Mat16x32, exactDiv(tileM, qmmaShape.m)> loadWholeCol() const
    {
        uint32_t const nbInstM = exactDiv(tileM, qmmaShape.m);
        Vec<Mat16x32, nbInstM> ret;
#pragma unroll
        for (uint32_t i = 0; i < nbInstM; i++)
        {
            ret[i] = load(i);
        }
        return ret;
    }

    __device__ inline LdGrain const* getPtr(uint32_t idxInstM) const
    {
        return checkedVal(basePtr + idxInstM * qmmaShape.m * srcCols, getPtrRef(idxInstM));
    }

private:
    __device__ inline LdGrain const* getPtrRef(uint32_t idxInstM) const
    {
        return &src.template at<true>(
            baseRow + idxInstM * qmmaShape.m + r, idxInstK * exactDiv(qmmaShape.k, grainElems) + c);
    }

    Src const& src;
    uint32_t const baseRow;
    uint32_t const idxInstK;
    uint32_t const r;
    uint32_t const c;
    LdGrain const* const basePtr;
};

using InstAcc = Array2D<float, 2, 2>;

using XBuffer = Array2D<LdGrain, headGrpSize, exactDiv(mathElemBytes* tokensPerTile, grainBytes)>;

struct CgaXBuffer
{
    XBuffer x;
    Vec<float, headGrpSize> rowSum;
};

struct PingPongMutex
{
    using ShmStorage = CtaBarrier[2];
    ShmStorage& barriers;
    uint32_t const idxGrp;

    static __device__ inline void initStorage(ShmStorage& barriers, uint32_t thrdsPerGrp)
    {
        new (&barriers[0]) CtaBarrier(thrdsPerGrp);
        new (&barriers[1]) CtaBarrier(thrdsPerGrp);
        barriers[0].arrive(thrdsPerGrp);
    }

    __device__ inline PingPongMutex(ShmStorage& shmStorage, uint32_t idxGrp)
        : barriers{shmStorage}
        , idxGrp{idxGrp}
    {
    }

    __device__ inline void lock(uint32_t iter)
    {
        barriers[idxGrp].wait_parity(toParity<1>(iter));
    }

    __device__ inline void unlock()
    {
        barriers[idxGrp ^ 1U].arrive();
    }
};

struct PartialResult
{
    static constexpr uint32_t nbChunks = 4;
    static constexpr uint32_t nbRowsPerChunk = exactDiv(headGrpSize, nbChunks);

    struct Chunk
    {
        Vec<OutputHead, nbRowsPerChunk> data;
        Vec<float, nbRowsPerChunk> rowSum;
        Vec<float, nbRowsPerChunk> rowMaxLog2e;
    };

    Chunk chunks[nbChunks];
};

constexpr uint32_t nbMathWarpsA = 8;
constexpr uint32_t nbComputeWarpsB = 8;
constexpr uint32_t nbMathGrpsA = 2;
constexpr uint32_t nbMathWarpsB = 8;

constexpr uint32_t nbMultiBlockBufs = 2;
constexpr uint32_t multiBlockMathWarps = 8;

constexpr bool useRegQ = USE_REG_Q;

struct SharedMemA
{
    static inline constexpr uint32_t nbKBufs = 4;
    static inline constexpr uint32_t nbXBufs = 1;

    static inline constexpr uint32_t regQParts = (useRegQ ? 1 : 0);
    static inline constexpr uint32_t shmQParts = nbQParts - regQParts;

    using ShmQPart = Array2D<LdGrain, headGrpSize, grainsPerPartK>;
    using ShmKPart = Array2D<LdGrain, tokensPerTile, grainsPerPartK>;

    Vec<ShmQPart, shmQParts> q;
    ShmKPart k[nbKBufs];
    XBuffer x[nbXBufs];
    Vec<float, headGrpSize> rowSum[nbXBufs];

    Vec<uint32_t, warp_size> drain; // data does not matter. Used to help avoid fence.

    // scaled by log2e. Write by last CGA iteration (from the other producer CTA) and read by current producer CTA.
    Vec<float, headGrpSize> rowMaxLog2e;
    // sync rowMaxLog2e between two producer CTAs and .consumed means the buffer for next iteration (in next producer)
    // is ready. The 4 groups from 2 producers CTAs form a ring
    CgaBarrier rowMaxLog2eBar[nbMathGrpsA];

    PingPongMutex::ShmStorage tensorCoreMutex;

    CtaBarrierPair kBars[nbKBufs];
    CtaBarrierPair xBars[nbXBufs];
#if USE_REG_Q
    CtaBarrierPair regQBar;
#endif
    CtaBarrier shmQBar;
    CgaBarrier cgaXBufConsumed;                    // for X

    PingPongMutex::ShmStorage rowMaxTransferMutex; // protect the order of rowMax transfer to consumers
    CgaBarrier consumerRowMaxConsumedBar;          // arrive by consumer CTAs.

    CtaBarrierPair multiBlockBars[nbMultiBlockBufs];

    __device__ inline void invalidateBarriers(uint32_t thrdIdx)
    {
        constexpr uint32_t nbBars = (useRegQ ? 15 : 13) + 2 * (nbKBufs + nbXBufs);
#ifndef __CUDACC_RTC__
        constexpr uint32_t nbBarsRef
            = exactDiv(offsetof(SharedMemA, qkScaleLog2e) - offsetof(SharedMemA, rowMaxLog2eBar), 8);
        static_assert(nbBars == nbBarsRef);
#endif
        if (thrdIdx < nbBars)
        {
            reinterpret_cast<CtaBarrier*>(&rowMaxLog2eBar[0])[thrdIdx].~CtaBarrier();
        }
    }

    __device__ inline Vec<PartialResult::Chunk, nbMultiBlockBufs>& getMultiBlockBufs()
    {
#ifndef __CUDACC_RTC__
        assert(sizeof(Vec<PartialResult::Chunk, nbMultiBlockBufs>) < offsetof(SharedMemA, rowMaxLog2eBar));
#endif
        return *reinterpret_cast<Vec<PartialResult::Chunk, nbMultiBlockBufs>*>(this);
    }

    float qkScaleLog2e;
    bool isLastSubSeq;
};

struct SharedMemB
{
    static inline constexpr uint32_t nbXVBufs = 2;
    static inline constexpr uint32_t nbXBufs = nbXVBufs;
    static inline constexpr uint32_t nbVBufs = nbXVBufs;

    using VBuffer
        = Vec<Array2D<LdGrain, tokensPerTile, exactDiv(partElemsV, grainElems)>, exactDiv(gemm1V, partElemsV)>;

    // x and v are using gemmK=128 per iteration. If we see high pressure on shared memory capacity, we can change to 64
    // in the future.
    struct XVBuffer
    {
        XBuffer x;
        VBuffer v;
        XBuffer pad; // for output swizzling
    };

    XVBuffer xv[nbXVBufs];

    __device__ inline XBuffer& x(uint32_t idx)
    {
        return xv[idx].x;
    }

    __device__ inline VBuffer& v(uint32_t idx)
    {
        return xv[idx].v;
    }

    Vec<float, headGrpSize> xRowSum[nbXBufs];

    static inline constexpr uint32_t nbAccRowMaxSumCopies = 2;
    Vec<float, headGrpSize> accRowMaxLog2e[nbAccRowMaxSumCopies];
    Vec<float, headGrpSize> accRowSum[nbAccRowMaxSumCopies];

    Vec<float, headGrpSize> xRowMaxLog2e[nbProducerCtasPerCga];
    CgaBarrier xRowMaxLog2eProducedBar[nbProducerCtasPerCga];

    CtaBarrierPair xBars[nbXBufs];
    CtaBarrierPair vBars[nbVBufs];

    CgaBarrier cgaXBufProduced[nbProducerCtasPerCga];
    CtaBarrier mathWarpsBar;

    CtaBarrierPair multiBlockBars[nbMultiBlockBufs];

    __device__ inline void invalidateBarriers(uint32_t thrdIdx)
    {
        constexpr uint32_t nbBars = 17;
#ifndef __CUDACC_RTC__
        constexpr uint32_t nbBarsRef
            = exactDiv(offsetof(SharedMemB, isLastSubSeq) - offsetof(SharedMemB, xRowMaxLog2eProducedBar), 8);
        static_assert(nbBars == nbBarsRef);
#endif
        if (thrdIdx < nbBars)
        {
            reinterpret_cast<CtaBarrier*>(&xRowMaxLog2eProducedBar[0])[thrdIdx].~CtaBarrier();
        }
    }

    __device__ inline Vec<PartialResult::Chunk, nbMultiBlockBufs>& getMultiBlockBufs()
    {
#ifndef __CUDACC_RTC__
        static_assert(
            sizeof(Vec<PartialResult::Chunk, nbMultiBlockBufs>) < offsetof(SharedMemB, xRowMaxLog2eProducedBar));
#endif
        return *reinterpret_cast<Vec<PartialResult::Chunk, nbMultiBlockBufs>*>(this);
    }

    bool isLastSubSeq;
};

__device__ void mergePartialOutputs(uint32_t& semaphore, Vec<OutputHead, PartialResult::nbRowsPerChunk>& dst,
    PartialResult const* reqPartialResults, uint32_t nbSubSeq, uint32_t ctaRank, uint32_t warpRank, uint2 warpIdx,
    void* sharedMem);

struct KernelArgs
{
    CUtensorMap const& tensorMapQ; // MhaIOHead[nbQHeads * totalNbInputTokens]
    CUtensorMap const& tensorMapK;
    CUtensorMap const& tensorMapV;
    float const& qScale;
    OutputHead* __restrict__ const& output; // [totalNbIntputTokens][nbQHeads]
    KVCacheList<usePagedKVCache> const& cacheList;
    uint32_t const& batchSize;
    float const* __restrict__ const&
        kvCacheScale; // Device memory scalar. Same scale for K and V cache. Used only for int8/fp8 KV cache.
    Vec<CgaXBuffer, nbProducerCtasPerCga>* __restrict__ const& cgaXBuf; // [totalNbInputTokens][maxNbSubSeq]
    uint32_t* __restrict__ const& semaphores;                           // [totalNbInputTokens]
    PartialResult* __restrict__ const& partialResults;                  // [totalNbInputTokens][maxNbSubSeq]
};

struct Producer
{
    static inline constexpr uint32_t nbMathGrps = nbMathGrpsA;
    static inline constexpr uint32_t nbMathWarps = nbMathWarpsA;
    static inline constexpr uint32_t nbMathThrds = nbMathWarps * warp_size;
    static inline constexpr uint32_t warpsPerGrp = exactDiv(nbMathWarps, nbMathGrps);
    static inline constexpr uint32_t thrdsPerGrp = warpsPerGrp * warp_size;
    static inline constexpr uint2 warpTile = {tokensPerTile, exactDiv(headGrpSize, warpsPerGrp)};
    using WarpAcc = WarpAccT<warpTile.y, warpTile.x>;
    using ThrdRegRowMax = ThrdRegRowMaxT<warpTile.y>;
    using QuadRegRowMax = QuadRegRowMaxT<warpTile.y>;

    KernelArgs const& args;
    SharedMemA& smem;
    uint32_t const maxNbSubSeq;
    uint32_t const idxReq;
    uint32_t const idxInputTokenGlobal;
    uint32_t const nbSubSeq;
    uint32_t const idxSubSeq;
    uint32_t const seqLen;
    uint32_t const ctaRank;
    uint32_t const warpRank;
    uint2 const warpIdx;

    __device__ inline Producer(KernelArgs const& args, SharedMemA& smem, uint32_t const maxNbSubSeq,
        uint32_t const idxReq, uint32_t idxInputTokenGlobal, uint32_t const seqLen, uint32_t const nbSubSeq,
        uint32_t const idxSubSeq, uint32_t ctaRank, uint32_t const warpRank, uint2 const warpIdx)
        : args(args)
        , smem(smem)
        , maxNbSubSeq(maxNbSubSeq)
        , idxReq(idxReq)
        , idxInputTokenGlobal(idxInputTokenGlobal)
        , seqLen(seqLen)
        , nbSubSeq(nbSubSeq)
        , idxSubSeq(idxSubSeq)
        , ctaRank(ctaRank)
        , warpRank(warpRank)
        , warpIdx(warpIdx)
    {
#ifndef NDEBUG
        if (threadIdx.x == 0)
        {
            asm("st.bulk.weak [%0], %1, 0;\n" ::"l"(&smem), "n"(sizeof(SharedMemA)) : "memory");
        }
        __syncthreads();
#endif
        if (threadIdx.x == 0)
        {
            smem.qkScaleLog2e = args.qScale * args.kvCacheScale[0] * log2e;
        }

        if (threadIdx.x < headGrpSize)
        {
            smem.rowMaxLog2e[threadIdx.x] = safeInitRowMax;
        }
        if (warpElectSync())
        {
            if (warpRank < SharedMemA::nbKBufs)
            {
                auto& b = smem.kBars[warpRank];
                b.initialize(1, thrdsPerGrp);
                b.consumed.arrive<Scope::CTA, ArriveOrder::RELAXED>(thrdsPerGrp);
            }
            if (warpRank < SharedMemA::nbXBufs)
            {
                auto& b = smem.xBars[warpRank];
                b.initialize(thrdsPerGrp, 1);
                b.consumed.arrive<Scope::CTA, ArriveOrder::RELAXED>(1);
            }
#if USE_REG_Q
            if (warpRank == 0)
            {
                smem.regQBar.initialize(1, nbMathThrds);
                smem.regQBar.consumed.arrive<Scope::CTA, ArriveOrder::RELAXED>(nbMathThrds);
            }
#endif
            if (warpRank < nbMathGrpsA)
            {
                auto& b = smem.rowMaxLog2eBar[warpRank];
                init(&b, thrdsPerGrp);
            }
            if (ctaRank == 0 && warpRank == 0)
            {
                smem.rowMaxLog2eBar[0].arrive<Scope::CTA, ArriveOrder::RELAXED>(thrdsPerGrp);
            }
            if (warpRank == 0)
            {
                init(&smem.shmQBar, 1);
                init(&smem.cgaXBufConsumed, 1 * nbVSplit);
                smem.cgaXBufConsumed.arrive<Scope::CTA, ArriveOrder::RELAXED>(1 * nbVSplit);
                PingPongMutex::initStorage(smem.tensorCoreMutex, thrdsPerGrp);
                PingPongMutex::initStorage(smem.rowMaxTransferMutex, thrdsPerGrp);
                init(&smem.consumerRowMaxConsumedBar, warp_size * nbComputeWarpsB * nbVSplit);
                smem.consumerRowMaxConsumedBar.arrive<Scope::CTA, ArriveOrder::RELAXED>(
                    warp_size * nbComputeWarpsB * nbVSplit);
            }
            if (nbSubSeq > 1 && warpRank < nbMultiBlockBufs)
            {
                auto& b = smem.multiBlockBars[warpRank];
                b.initialize(1, warp_size * multiBlockMathWarps);
                b.consumed.arrive<Scope::CTA, ArriveOrder::RELAXED>(warp_size * multiBlockMathWarps);
            }
        }
        clusterBarArrive();
        clusterBarWait();
    }

    __device__ inline ~Producer()
    {
        clusterBarArrive();
        clusterBarWait();
        smem.invalidateBarriers(threadIdx.x);
    }

    __device__ inline void run()
    {
        if (warpIdx.y == 2)
        { // IO warps
            asm volatile("setmaxnreg.dec.sync.aligned.u32 %0;\n" ::"n"(nbRegsForIOWarps));
            if (warpIdx.x == 0)
            { // q
                loadQ();
            }
            else if (warpIdx.x == 1)
            { // k
                loadK();
            }
            else if (warpIdx.x == 2)
            { // x
                sendX();
            }
        }
        else
        { // Compute warps
            asm volatile("setmaxnreg.inc.sync.aligned.u32 %0;\n" ::"n"(nbRegsForMathWarps));
            compute();
        }
        if (nbSubSeq > 1)
        {
            mergePartialOutputs(args.semaphores[idxInputTokenGlobal],
                reinterpret_cast<Vec<OutputHead, PartialResult::nbRowsPerChunk>&>(
                    args.output[headGrpSize * idxInputTokenGlobal + PartialResult::nbRowsPerChunk * ctaRank]),
                args.partialResults + maxNbSubSeq * idxInputTokenGlobal, nbSubSeq, ctaRank, warpRank, warpIdx, &smem);
        }
    }

private:
    __device__ inline uint32_t iterStride() const
    {
        return nbSubSeq * nbProducerCtasPerCga;
    }

    __device__ inline uint32_t idxTileBeg() const
    {
        return nbProducerCtasPerCga * idxSubSeq + ctaRank;
    }

    __device__ inline uint32_t nbTiles() const
    {
        return divUp(seqLen, tokensPerTile);
    }

    __device__ inline SharedMemB& getConsumerShm(uint32_t const idxConsumer)
    {
        return *mapa(reinterpret_cast<SharedMemB*>(&smem), nbProducerCtasPerCga + idxConsumer);
    };

    static constexpr uint32_t regQPartShmBeg = SharedMemA::shmQParts - SharedMemA::regQParts;

    __device__ inline void loadQ()
    {
#if USE_REG_Q
        static_assert(SharedMemA::regQParts <= SharedMemA::shmQParts);
        smem.regQBar.consumed.wait_parity(toParity<1>(0));
#pragma unroll 1
        for (uint32_t i = 0; i < SharedMemA::regQParts; i++)
        {
            if (warpElectSync())
            {
                tma::loadAsync(&smem.q[regQPartShmBeg + i], args.tensorMapQ,
                    DimsLE<2>{partElemsK * i, headGrpSize * idxInputTokenGlobal}, smem.regQBar.produced);
            }
        }
        if (warpElectSync())
        {
            smem.regQBar.produced.arrive_tx(sizeof(SharedMemA::ShmQPart) * SharedMemA::regQParts);
        }
#endif
#pragma unroll 1
        for (uint32_t i = 0; i < SharedMemA::shmQParts; i++)
        {
            uint32_t const idxPart = SharedMemA::regQParts + i;
#if USE_REG_Q
            if (i == regQPartShmBeg)
            {
                smem.regQBar.consumed.wait_parity(toParity<1>(1));
            }
#endif
            if (warpElectSync())
            {
                tma::loadAsync(&smem.q[i], args.tensorMapQ,
                    DimsLE<2>{partElemsK * idxPart, headGrpSize * idxInputTokenGlobal}, smem.shmQBar);
            }
        }
        if (warpElectSync())
        {
            smem.shmQBar.arrive_tx(sizeof(SharedMemA::ShmQPart) * SharedMemA::shmQParts);
        }
    }

    __device__ inline void loadK();

    __device__ inline void sendX();

    __device__ inline void compute()
    {
        uint32_t const grpIdx = warpIdx.y;
        uint32_t const tileBaseRow = warpTile.y * warpIdx.x;
        PingPongMutex tensorCoreMutex{smem.tensorCoreMutex, grpIdx};
        PingPongMutex rowMaxTransferMutex{smem.rowMaxTransferMutex, grpIdx};

        constexpr uint32_t partNbInstK = exactDiv(partElemsK, qmmaShape.k);
        using AtomA = Vec<uint32_t, 4>; // for 16x32 data, working as mat A of QMMA.16832
        using RegQPartCol = Vec<AtomA, exactDiv(warpTile.y, qmmaShape.m)>;
        using RegQPart = Vec<RegQPartCol, partNbInstK>;
        using RegQ = Vec<RegQPart, SharedMemA::regQParts>;
        constexpr uint32_t tileNbAtomBx2 = exactDiv(tokensPerTile, qmmaShape.n * 2);
        using AtomBx2 = Vec<uint32_t, 4>; // one AtomB is 8x32 and AtomBx2 is 16x32
        using RegKPartCol = Vec<AtomBx2, tileNbAtomBx2>;
        using RegKPart = Vec<RegKPartCol, partNbInstK>;

        uint32_t const lane = laneId();
        uint32_t const rA = lane % 16;
        uint32_t const cA = lane / 16;
        uint32_t const rB = (lane / 16) * 8 + lane % 8;
        uint32_t const cB = (lane % 16) / 8;
        auto loadRegQCol = [&](SharedMemA::ShmQPart const& q, uint32_t idxInstK) -> RegQPartCol
        {
            Mat16x32Loader const loaderQ(q, tileBaseRow, idxInstK, rA, cA);
            return loaderQ.loadWholeCol<warpTile.y>();
        };
        auto loadRegKCol = [&](SharedMemA::ShmKPart const& k, uint32_t idxInstK) -> RegKPartCol
        {
            Mat16x32Loader const loaderK(k, 0, idxInstK, rB, cB);
            return loaderK.loadWholeCol<warpTile.x>();
        };
        auto loadPart = [&](auto const& loadCol, auto const& shmPart)
        {
            mha::conditional_t<mha::is_same_v<SharedMemA::ShmQPart, mha::decay_t<decltype(shmPart)>>, RegQPart,
                RegKPart>
                regPart;
#pragma unroll
            for (uint32_t idxInstK = 0; idxInstK < partNbInstK; idxInstK++)
            {
                regPart[idxInstK] = loadCol(shmPart, idxInstK);
            }
            return regPart;
        };

#if USE_REG_Q
        // load regQ
        smem.regQBar.produced.wait_parity(toParity<1>(0));
        RegQ regQ;
#pragma unroll
        for (uint32_t idxPart = 0; idxPart < SharedMemA::regQParts; idxPart++)
        {
            uint32_t const idxBuf = regQPartShmBeg + idxPart;
            regQ[idxPart] = loadPart(loadRegQCol, smem.q[idxBuf]);
        }
        smem.regQBar.consumed.arrive();
#endif
        smem.shmQBar.wait_parity(false);
// main loop
#pragma unroll 1
        for (uint32_t grpIter = 0; true; grpIter++)
        {
            uint32_t const ctaIter = grpIdx + grpIter * nbMathGrps;
            uint32_t const idxTile = idxTileBeg() + iterStride() * ctaIter;
            if (idxTile >= nbTiles())
            {
                break;
            }
            WarpAcc acc{};
            // wait until it's our turn
            tensorCoreMutex.lock(grpIter);
            BarWaiter kBarWaiter(smem.kBars, ctaIter * nbKParts + 0);
            kBarWaiter.testWait();
            RegQPart regQBuf;
#if USE_REG_Q
            static_assert(SharedMemA::regQParts > 0);
            regQBuf[0] = regQ[0][0];
#else
            regQBuf[0] = loadRegQCol(smem.q[0], 0);
#endif
            kBarWaiter.wait();
            RegKPart regKBuf;
            regKBuf[0] = loadRegKCol(smem.k[kBarWaiter.idxBuf], 0);

            auto shouldTestWait = [](uint32_t idxInstK, uint32_t idxAtomBx2)
            { return idxInstK == partNbInstK - 1 && idxAtomBx2 == tileNbAtomBx2 - 2; };
            BarWaiter kBarWaiterNext = kBarWaiter.next();
#if USE_REG_Q
#pragma unroll
            for (uint32_t idxPart = 0; idxPart < SharedMemA::regQParts; idxPart++)
            {
#pragma unroll
                for (uint32_t idxInstK = 0; idxInstK < partNbInstK; idxInstK++)
                {
                    bool const prefetchNextPart = (idxInstK == partNbInstK - 1);
                    uint32_t const idxPartPrefetch = prefetchNextPart ? idxPart + 1 : idxPart;
                    uint32_t const idxInstKPrefetch = prefetchNextPart ? 0 : idxInstK + 1;
                    bool const prefetch = (!prefetchNextPart || (idxPart < nbKParts - 1));

                    if (prefetchNextPart)
                    {
                        kBarWaiter = kBarWaiterNext;
                        kBarWaiterNext = kBarWaiter.next();
                        if (prefetch)
                        {
                            kBarWaiter.wait();
                        }
                    }

                    Mat16x32Loader const loaderK(smem.k[kBarWaiter.idxBuf], 0, idxInstKPrefetch, rB, cB);
#pragma unroll
                    for (uint32_t idxAtomBx2 = 0; idxAtomBx2 < tileNbAtomBx2; idxAtomBx2++)
                    {
                        if (idxAtomBx2 == 2 && prefetch)
                        {
                            if (idxPartPrefetch < SharedMemA::regQParts)
                            {
                                regQBuf[idxInstKPrefetch] = regQ[idxPartPrefetch][idxInstKPrefetch];
                            }
                            else
                            {
                                regQBuf[idxInstKPrefetch]
                                    = loadRegQCol(smem.q[idxPartPrefetch - SharedMemA::regQParts], idxInstKPrefetch);
                            }
                        }
                        AtomBx2 const& atomBx2 = regKBuf[idxInstK][idxAtomBx2];
                        regKBuf[idxInstKPrefetch][idxAtomBx2] = loaderK.load(idxAtomBx2);
                        if (shouldTestWait(idxInstKPrefetch, idxAtomBx2) && prefetch)
                        {
                            kBarWaiterNext.testWait();
                        }
#pragma unroll
                        for (uint32_t i = 0; i < WarpAcc::rows; i++)
                        {
#pragma unroll
                            for (uint32_t j = 0; j < 2; j++)
                            {
                                mma<__hip_fp8_e4m3_fnuz>(reinterpret_cast<float(&)[2][2]>(acc(i, 2 * idxAtomBx2 + j)),
                                    reinterpret_cast<uint32_t const(&)[2][2]>(regQBuf[idxInstK][i]),
                                    reinterpret_cast<uint32_t const(&)[2][1]>(atomBx2[2 * j]));
                            }
                        }
                        if (prefetch)
                        {
                            regKBuf[idxInstKPrefetch][idxAtomBx2] = loaderK.load(idxAtomBx2);
                        }
                    }
                    if (idxInstKPrefetch == partNbInstK - 1)
                    {
                        assert(prefetch);
                        kBarWaiter.consumed();
                    }
                }
            }
#endif
#pragma unroll
            for (uint32_t idxPart = SharedMemA::regQParts; idxPart < nbQParts; idxPart++)
            {
#pragma unroll
                for (uint32_t idxInstK = 0; idxInstK < partNbInstK; idxInstK++)
                {
                    bool const prefetchNextPart = (idxInstK == partNbInstK - 1);
                    uint32_t const idxPartPrefetch = prefetchNextPart ? idxPart + 1 : idxPart;
                    uint32_t const idxInstKPrefetch = prefetchNextPart ? 0 : idxInstK + 1;
                    bool const prefetch = (!prefetchNextPart || (idxPart < nbKParts - 1));

                    if (prefetchNextPart)
                    {
                        kBarWaiter = kBarWaiterNext;
                        kBarWaiterNext = kBarWaiter.next();
                        if (prefetch)
                        {
                            kBarWaiter.wait();
                        }
                    }

                    Mat16x32Loader const loaderK(smem.k[kBarWaiter.idxBuf], 0, idxInstKPrefetch, rB, cB);
#pragma unroll
                    for (uint32_t idxAtomBx2 = 0; idxAtomBx2 < tileNbAtomBx2; idxAtomBx2++)
                    {
                        if (idxAtomBx2 == 2 && prefetch)
                        {
                            regQBuf[idxInstKPrefetch]
                                = loadRegQCol(smem.q[idxPartPrefetch - SharedMemA::regQParts], idxInstKPrefetch);
                        }
                        AtomBx2 const& atomBx2 = regKBuf[idxInstK][idxAtomBx2];
                        if (shouldTestWait(idxInstKPrefetch, idxAtomBx2) && prefetch)
                        {
                            kBarWaiterNext.testWait();
                        }
#pragma unroll
                        for (uint32_t i = 0; i < WarpAcc::rows; i++)
                        {
#pragma unroll
                            for (uint32_t j = 0; j < 2; j++)
                            {
                                mma<__hip_fp8_e4m3_fnuz>(reinterpret_cast<float(&)[2][2]>(acc(i, 2 * idxAtomBx2 + j)),
                                    reinterpret_cast<uint32_t const(&)[2][2]>(regQBuf[idxInstK][i]),
                                    reinterpret_cast<uint32_t const(&)[2][1]>(atomBx2[2 * j]));
                            }
                        }
                        if (prefetch)
                        {
                            regKBuf[idxInstKPrefetch][idxAtomBx2] = loaderK.load(idxAtomBx2);
                        }
                    }
                    if (idxInstKPrefetch == partNbInstK - 1)
                    {
                        assert(prefetch);
                        kBarWaiter.consumed();
                        if (idxPartPrefetch == nbKParts - 1)
                        {
                            tensorCoreMutex.unlock(); // let the other group to use tensor cores
                        }
                    }
                }
            }
            uint32_t const validTokens = seqLen - tokensPerTile * idxTile;
            if (validTokens < tokensPerTile)
            {
                applyMask(this_warp(), acc, 0, validTokens);
            }
            WarpAcc const xF32 = scaleAndSoftmax(acc, grpIdx, grpIter, tileBaseRow, rowMaxTransferMutex);
            // convert to fp8
            WarpAcc const xF32Quant = xF32 * rcpXScale;
            // 0, 1, 8, 9,  2, 3, 10, 11,  4, 5, 12, 13,  6, 7, 14, 15
            Array2D<Array2D<uint32_t, 2, 1>, WarpAcc::rows, exactDiv(WarpAcc::cols, 2)> xF8;
#pragma unroll
            for (uint32_t i = 0; i < WarpAcc::rows; i++)
            {
#pragma unroll
                for (uint32_t m = 0; m < exactDiv(qmmaShape.m, 8); m++)
                {
#pragma unroll
                    for (uint32_t j = 0; j < WarpAcc::cols; j += 2)
                    {
                        auto& dst = reinterpret_cast<__hip_fp8x2_e4m3_fnuz(&)[2]>(xF8(i, j / 2)(m, 0));
                        dst[0] = __hip_fp8x2_e4m3_fnuz(float2{xF32Quant(i, j)(m, 0), xF32Quant(i, j)(m, 1)});
                        dst[1] = __hip_fp8x2_e4m3_fnuz(float2{xF32Quant(i, j + 1)(m, 0), xF32Quant(i, j + 1)(m, 1)});
                    }
                }
            }
            // use tensor core to compute rowSum
            ThrdRegRowMax const rowSum = computeRowSumFromF8
                ? computeRowSumF8<warpTile.y, warpTile.x>(this_warp(), xF8)
                : computeRowSumF32<warpTile.y, warpTile.x>(this_warp(), xF32);

            // store xF8 and rowSum into L2 scratch buffer
            uint32_t const idxXBuf = ctaIter % SharedMemA::nbXBufs;
            auto& xBar = smem.xBars[idxXBuf];
            xBar.consumed.wait_parity(toParity<SharedMemA::nbXBufs>(ctaIter));
            storeRowMax<warpTile.y>(smem.rowSum[idxXBuf], rowSum, tileBaseRow, lane);
            storeOrderedXToShm(smem.x[idxXBuf], xF8, tileBaseRow, lane);
            xBar.produced.arrive();
        }
    }

    __device__ inline WarpAcc scaleAndSoftmax(WarpAcc const& acc, uint32_t grpIdx, uint32_t grpIter,
        uint32_t tileBaseRow, PingPongMutex& rowMaxTransferMutex);

    __device__ inline void storeOrderedXToShm(XBuffer& dst,
        Array2D<Array2D<uint32_t, 2, 1>, WarpAcc::rows, exactDiv(WarpAcc::cols, 2)> const& src,
        uint32_t const tileBaseRow, uint32_t const lane = laneId());
};

__device__ inline void Producer::loadK()
{
    KVTilePartLoader loader
    {
        args.cacheList, idxReq, args.tensorMapK
#if USE_PAGED_KV_CACHE
            ,
            divUp(seqLen, tokensPerPage)
#endif
    };

#pragma unroll 1
    for (uint32_t iter = 0; true; iter++)
    {
        uint32_t const idxTile = idxTileBeg() + iterStride() * iter;
        if (idxTile >= nbTiles())
        {
            break;
        }
        uint32_t const idxPageBuf = iter % KVTilePartLoader::nbPageBuffers;
        loader.loadPages(idxTile, idxPageBuf);
#pragma unroll 1
        for (uint32_t idxPart = 0; idxPart < nbKParts; idxPart++)
        {
            uint32_t const idxPartGlobal = iter * nbKParts + idxPart;
            uint32_t const idxBuf = idxPartGlobal % SharedMemA::nbKBufs;
            auto& bar = smem.kBars[idxBuf];
            bar.consumed.wait_parity(toParity<SharedMemA::nbKBufs>(idxPartGlobal));
            loader.loadData(smem.k[idxBuf], idxTile, partElemsK * idxPart, bar.produced, idxPageBuf);
            if (warpElectSync())
            {
                bar.produced.arrive_tx(sizeof(SharedMemA::ShmKPart));
            }
        }
    }
}

__device__ inline void Producer::sendX()
{
    for (uint32_t iter = 0; true; iter++)
    {
        uint32_t const idxTile = idxTileBeg() + iterStride() * iter;
        if (idxTile >= nbTiles())
        {
            break;
        }
        uint32_t const idxBuf = iter % SharedMemA::nbXBufs;
        auto& xBar = smem.xBars[idxBuf];
        xBar.produced.wait_parity(toParity<SharedMemA::nbXBufs>(iter));
        smem.cgaXBufConsumed.wait_parity(toParity<1>(iter));
        if (warpElectSync())
        {
            auto& dst = args.cgaXBuf[nbSubSeq * idxInputTokenGlobal + idxSubSeq][ctaRank];
            tma::store1DAsync(&dst.x, &smem.x[idxBuf], sizeof(XBuffer));
            tma::store1DAsync(&dst.rowSum, &smem.rowSum[idxBuf], sizeof(smem.rowSum[0]));
            tma::commitGroup();
            tma::waitGroup<0>();
            xBar.consumed.arrive();
            asm volatile("fence.release.cluster;\n");
#pragma unroll
            for (uint32_t i = 0; i < nbVSplit; i++)
            {
                auto& producedBar = getConsumerShm(i).cgaXBufProduced[ctaRank];
                producedBar.arrive<Scope::CGA, ArriveOrder::RELAXED>();
            }
        }
    }
}

__device__ inline Producer::WarpAcc Producer::scaleAndSoftmax(
    WarpAcc const& acc, uint32_t grpIdx, uint32_t grpIter, uint32_t tileBaseRow, PingPongMutex& rowMaxTransferMutex)
{
    uint32_t const ctaIter = grpIdx + grpIter * nbMathGrps;
    uint32_t const cgaIter = ctaRank + ctaIter * nbProducerCtasPerCga;
    auto const warp = this_warp();
    uint32_t const lane = laneId();
    uint32_t const idxProducer = ctaRank;
    assert(ctaRank < nbProducerCtasPerCga);

    auto const accLog2e = acc * smem.qkScaleLog2e;
    bool const skipWaitLastShmRowMax = smem.rowMaxLog2eBar[grpIdx].test_wait_parity(toParity<1>(grpIter));
    QuadRegRowMax const tileRowMaxLog2e = computeRowMax<warpTile.y, warpTile.x>(accLog2e);
    // get max with previous CTA's rowMax
    if (!skipWaitLastShmRowMax)
    {
        smem.rowMaxLog2eBar[grpIdx].wait_parity(toParity<1>(grpIter));
    }
    auto const lastRowMaxLog2e = loadShmRowMax<warpTile.y>(smem.rowMaxLog2e, tileBaseRow, lane);

    auto const quadRowMaxLog2e = fmaxf(tileRowMaxLog2e, replicateForQuad(warp, lastRowMaxLog2e));

    // transfer new row max to the other producer CTA for next iteration
    SharedMemA& smemNext = mapa(smem, ctaRank ^ 1U);
    CgaBarrier& nextRowMaxLog2eBar
        = smemNext.rowMaxLog2eBar[(cgaIter + 1) % (nbMathGrps * nbProducerCtasPerCga) / nbMathGrps];
    ThrdRegRowMax const rowMaxLog2e = dedupFromQuad(warp, quadRowMaxLog2e);
    storeRowMaxAsync<warpTile.y>(nextRowMaxLog2eBar, smemNext.rowMaxLog2e, rowMaxLog2e, tileBaseRow, lane);
    nextRowMaxLog2eBar.arrive_tx_relaxed(sizeof(rowMaxLog2e)); // notify that the next CTA can read rowMax now.

    // transfer rowMax to consumers.
    rowMaxTransferMutex.lock(grpIter); // @fixme: use test_wait_parity() early to avoid latency.
    smem.consumerRowMaxConsumedBar.wait_parity(checkedVal<bool>(grpIdx, toParity<1>(ctaIter)));
    for (uint32_t idxConsumer = 0; idxConsumer < nbVSplit; idxConsumer++)
    {
        auto& smemB = getConsumerShm(idxConsumer);
        storeRowMaxAsync<warpTile.y>(smemB.xRowMaxLog2eProducedBar[idxProducer], smemB.xRowMaxLog2e[idxProducer],
            rowMaxLog2e, tileBaseRow, lane);
        smemB.xRowMaxLog2eProducedBar[idxProducer].arrive_tx_relaxed(sizeof(rowMaxLog2e));
    }
    rowMaxTransferMutex.unlock();

    WarpAcc x;
// apply softmax
#pragma unroll
    for (uint32_t m = 0; m < acc.rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
            float const maxVal = quadRowMaxLog2e[m * InstAcc::rows + i];
#pragma unroll
            for (uint32_t n = 0; n < acc.cols; n++)
            {
#pragma unroll
                for (uint32_t j = 0; j < InstAcc::cols; j++)
                {
                    float elem = accLog2e(m, n)(i, j);
                    assert(maxVal >= elem);
                    x(m, n)(i, j) = exp2f(elem - maxVal);
                }
            }
        }
    }

    return x;
}

__device__ inline void Producer::storeOrderedXToShm(XBuffer& dst,
    Array2D<Array2D<uint32_t, 2, 1>, WarpAcc::rows, exactDiv(WarpAcc::cols, 2)> const& src, uint32_t const tileBaseRow,
    uint32_t const lane)
{
    uint32_t const r = lane % 16;
    uint32_t const c = lane / 16;
    using Src = mha::decay_t<decltype(src)>;
    LdGrain* ptrs[exactDiv(Src::cols, 2)][Src::rows];
#pragma unroll
    for (uint32_t idxInstK = 0; idxInstK < exactDiv(Src::cols, 2); idxInstK++)
    {
        Mat16x32Loader const loader(dst, tileBaseRow, idxInstK, r, c);
#pragma unroll
        for (uint32_t idxInstM = 0; idxInstM < Src::rows; idxInstM++)
        {
            auto const p = const_cast<LdGrain*>(loader.getPtr(idxInstM));
            stmatrix<false, 4>(p, reinterpret_cast<LdGrain const&>(src(idxInstM, idxInstK * 2)));
            ptrs[idxInstK][idxInstM] = p;
        }
    }
    // reorder from 0, 1, 8, 9,  2, 3, 10, 11,  4, 5, 12, 13,  6, 7, 14, 15
    // to 0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15
    __syncwarp();
#pragma unroll
    for (uint32_t idxInstK = 0; idxInstK < exactDiv(Src::cols, 2); idxInstK++)
    {
#pragma unroll
        for (uint32_t idxInstM = 0; idxInstM < Src::rows; idxInstM++)
        {
            auto const p = ptrs[idxInstK][idxInstM];
            auto const i = *p;
            LdGrain const o = {prmt(i[0], i[1], PermuteOrder{0, 1, 4, 5}), prmt(i[2], i[3], PermuteOrder{0, 1, 4, 5}),
                prmt(i[0], i[1], PermuteOrder{2, 3, 6, 7}), prmt(i[2], i[3], PermuteOrder{2, 3, 6, 7})};
            *p = o;
        }
    }
}

struct Consumer
{
    static inline constexpr uint32_t nbMathWarps = nbMathWarpsB;
    static inline constexpr uint32_t nbMathThrds = warp_size * nbMathWarps;
    static inline constexpr uint2 ctaShape = {2, 4};
    static_assert(SharedMemB::nbAccRowMaxSumCopies == ctaShape.x);
    static_assert(ctaShape.x * ctaShape.y == nbMathWarps);
    static inline constexpr uint2 warpTile = {exactDiv(gemm1V, ctaShape.x), exactDiv(headGrpSize, ctaShape.y)};

    static inline constexpr uint32_t nbWarpOutSwizzleBuf = nbMathWarps;
    using WarpOutSwizzleBuf = Array2D<LdGrain,
        exactDiv(exactDiv(sizeof(SharedMemB::XVBuffer), sizeof(OutputElem) * warpTile.x), nbMathWarps),
        exactDiv(sizeof(OutputElem) * warpTile.x, grainBytes)>;
    static_assert(WarpOutSwizzleBuf::rows % 8 == 0);

    using WarpAcc = WarpAccT<warpTile.y, warpTile.x>;
    using ThrdRegRowMax = ThrdRegRowMaxT<warpTile.y>;
    using UniformNeedRescaleMask = Vec<uint32_t, divUp(warpTile.y, warp_size)>;

    KernelArgs const& args;
    SharedMemB& smem;
    uint32_t const maxNbSubSeq;
    uint32_t const idxReq;
    uint32_t const idxInputTokenGlobal;
    uint32_t const nbSubSeq;
    uint32_t const idxSubSeq;
    uint32_t const seqLen;
    uint32_t const ctaRank;
    uint32_t const warpRank;
    uint2 const warpIdx;

    __device__ inline uint32_t iterStride() const
    {
        return nbSubSeq * nbProducerCtasPerCga;
    }

    __device__ inline uint32_t idxTileBeg() const
    {
        return nbProducerCtasPerCga * idxSubSeq;
    }

    __device__ inline uint32_t nbTiles() const
    {
        return divUp(seqLen, tokensPerTile);
    }

    __device__ inline uint32_t idxConsumer() const
    {
        return ctaRank - 2;
    }

    __device__ inline Consumer(KernelArgs const& args, SharedMemB& smem, uint32_t const maxNbSubSeq,
        uint32_t const idxReq, uint32_t const idxInputTokenGlobal, uint32_t const seqLen, uint32_t const nbSubSeq,
        uint32_t const idxSubSeq, uint32_t ctaRank, uint32_t const warpRank, uint2 const warpIdx)
        : args(args)
        , smem(smem)
        , maxNbSubSeq(maxNbSubSeq)
        , idxReq(idxReq)
        , idxInputTokenGlobal(idxInputTokenGlobal)
        , seqLen(seqLen)
        , nbSubSeq(nbSubSeq)
        , idxSubSeq(idxSubSeq)
        , ctaRank(ctaRank)
        , warpRank(warpRank)
        , warpIdx(warpIdx)
    {
#ifndef NDEBUG
        if (threadIdx.x == 0)
        {
            asm("st.bulk.weak [%0], %1, 0;\n" ::"l"(&smem), "n"(sizeof(SharedMemB)) : "memory");
        }
        __syncthreads();
#endif
        if (threadIdx.x < headGrpSize)
        {
            for (uint32_t i = 0; i < SharedMemB::nbAccRowMaxSumCopies; i++)
            {
                smem.accRowMaxLog2e[i][threadIdx.x] = safeInitRowMax;
                smem.accRowSum[i][threadIdx.x] = 0;
            }
        }
        if (warpElectSync())
        {
            if (warpRank < nbProducerCtasPerCga)
            {
                init(&smem.xRowMaxLog2eProducedBar[warpRank], Producer::thrdsPerGrp);
                init(&smem.cgaXBufProduced[warpRank], 1);
            }
            if (warpRank < SharedMemB::nbXBufs)
            {
                auto& bar = smem.xBars[warpRank];
                bar.initialize(1, nbMathThrds);
                bar.consumed.arrive<Scope::CTA, ArriveOrder::RELAXED>(nbMathThrds);
            }
            if (warpRank < SharedMemB::nbVBufs)
            {
                auto& bar = smem.vBars[warpRank];
                bar.initialize(1, nbMathThrds);
                bar.consumed.arrive<Scope::CTA, ArriveOrder::RELAXED>(nbMathThrds);
            }
            if (warpRank == 0)
            {
                init(&smem.mathWarpsBar, warp_size * nbMathWarps);
            }
            if (nbSubSeq > 1 && warpRank < nbMultiBlockBufs)
            {
                auto& b = smem.multiBlockBars[warpRank];
                b.initialize(1, warp_size * multiBlockMathWarps);
                b.consumed.arrive<Scope::CTA, ArriveOrder::RELAXED>(warp_size * multiBlockMathWarps);
            }
        }
        clusterBarArrive();
        clusterBarWait();
    }

    __device__ inline ~Consumer()
    {
        clusterBarArrive();
        clusterBarWait();
        smem.invalidateBarriers(threadIdx.x);
    }

    __device__ inline void run()
    {
        if (warpIdx.y == 2)
        {
            asm volatile("setmaxnreg.dec.sync.aligned.u32 %0;\n" ::"n"(nbRegsForIOWarps));
            if (warpIdx.x == 0)
            {
                loadX();
            }
            else if (warpIdx.x == 1)
            {
                loadV();
            }
        }
        else
        {
            asm volatile("setmaxnreg.inc.sync.aligned.u32 %0;\n" ::"n"(nbRegsForMathWarps));
            compute();
        }
        if (nbSubSeq > 1)
        {
            mergePartialOutputs(args.semaphores[idxInputTokenGlobal],
                reinterpret_cast<Vec<OutputHead, PartialResult::nbRowsPerChunk>&>(
                    args.output[headGrpSize * idxInputTokenGlobal + PartialResult::nbRowsPerChunk * ctaRank]),
                args.partialResults + maxNbSubSeq * idxInputTokenGlobal, nbSubSeq, ctaRank, warpRank, warpIdx, &smem);
        }
    }

    __device__ inline void loadX();
    __device__ inline void loadV();
    __device__ inline void compute();

    __device__ inline uint32_t iterToTile(uint32_t iter) const
    {
        return idxTileBeg() + iterStride() * (iter / 2) + iter % 2;
    }

    __device__ inline SharedMemA& getProducerShm(uint32_t idxProducer) const
    {
        return mapa(reinterpret_cast<SharedMemA&>(smem), idxProducer);
    }

    using WarpOutputTile = Array2D<uint32_t, InstAcc::rows * Consumer::WarpAcc::rows, Consumer::WarpAcc::cols>;
    __device__ inline WarpOutputTile finalize(
        WarpAcc const& acc, ThrdRegRowMax const& accRowSum, float xvScale, uint32_t lane = laneId());
    __device__ inline void storeOutput(Vec<OutputHead, warpTile.y>& dst, uint32_t dstBaseCol,
        WarpOutputTile const& regTile, WarpOutSwizzleBuf& swizzleBuf, uint32_t lane = laneId());
};

__device__ inline void Consumer::compute()
{
    uint2 const tileIdx = {warpIdx.y, warpIdx.x};
    uint2 const tileBase = {tileIdx.x * warpTile.x, tileIdx.y * warpTile.y};

    constexpr uint32_t tileNbInstK = exactDiv(tokensPerTile, qmmaShape.k);
    constexpr uint32_t warpTileNbAtomBx2 = exactDiv(warpTile.x, qmmaShape.n * 2);

    uint32_t const lane = laneId();
    uint32_t const idxHalf = lane / 16;
    uint32_t const laneInHalf = lane % 16;
    uint32_t const rA = laneInHalf;
    uint32_t const cA = idxHalf;
    uint32_t const rB = lane;
    uint32_t const cB = 0;

    WarpAcc acc{};
    uint32_t idxXVBufLast;
    bool skipWait_xRowMaxLog2eProducedBar = false;
    for (uint32_t iter = 0; true; iter++)
    {
        uint32_t const idxTile = iterToTile(iter);
        if (idxTile >= nbTiles())
        {
            break;
        }

        ThrdRegRowMax accRowMaxLog2e = loadShmRowMax<warpTile.y>(smem.accRowMaxLog2e[tileIdx.x], tileBase.y, lane);
        ThrdRegRowMax accRowSum = loadShmRowMax<warpTile.y>(smem.accRowSum[tileIdx.x], tileBase.y, lane);

        uint32_t const idxProducer = iter % nbProducerCtasPerCga;
        if (!skipWait_xRowMaxLog2eProducedBar)
        {
            smem.xRowMaxLog2eProducedBar[idxProducer].wait_parity(toParity<nbProducerCtasPerCga>(iter));
        }
        ThrdRegRowMax const xRowMaxLog2e = loadShmRowMax<warpTile.y>(smem.xRowMaxLog2e[idxProducer], tileBase.y, lane);
        auto& prodSmem = getProducerShm(idxProducer);
        uint32_t const drainData = hashRegData(xRowMaxLog2e);
        tma::storeAsync(&prodSmem.drain[lane], drainData, prodSmem.consumerRowMaxConsumedBar);
        prodSmem.consumerRowMaxConsumedBar.template arrive_tx<Scope::CGA, ArriveOrder::RELAXED>(sizeof(drainData));
        assert(all(accRowMaxLog2e <= xRowMaxLog2e));

        auto const needRescaleVec = (xRowMaxLog2e > accRowMaxLog2e);
        UniformNeedRescaleMask rescaleMask;
#pragma unroll
        for (uint32_t i = 0; i < rescaleMask.size; i++)
        {
            rescaleMask[i] = __ballot_sync(~0U, needRescaleVec[i]);
        }
        bool const anyNeedRescale = any(rescaleMask != UniformNeedRescaleMask::filled(0));
        if (anyNeedRescale)
        {
            auto const scaleVec = exp2f(accRowMaxLog2e - xRowMaxLog2e);
#pragma unroll
            for (uint32_t m = 0; m < WarpAcc::rows; m++)
            {
#pragma unroll
                for (uint32_t i = 0; i < InstAcc::rows; i++)
                {
                    uint8_t const mask = reinterpret_cast<uint8_t const(&)[2][2]>(rescaleMask[m / 2])[m % 2][i];
                    bool const needRescale = (mask != 0);
                    if (needRescale)
                    { // this branch is warp-uniform
                        float const scale = __shfl_sync(~0U, scaleVec[m / 2], 16 * (m % 2) + 8 * i + lane / 4);
#pragma unroll
                        for (uint32_t n = 0; n < WarpAcc::cols; n++)
                        {
#pragma unroll
                            for (uint32_t j = 0; j < InstAcc::cols; j++)
                            {
                                acc(m, n)(i, j) *= scale;
                            }
                        }
                    }
                }
            }
            accRowSum = accRowSum * scaleVec;
        }
        accRowMaxLog2e = xRowMaxLog2e;
        storeRowMax<warpTile.y>(smem.accRowMaxLog2e[tileIdx.x], accRowMaxLog2e, tileBase.y, lane);

        uint32_t const idxXBuf = iter % SharedMemB::nbXBufs;
        uint32_t const idxVBuf = iter % SharedMemB::nbVBufs;
        auto& xBar = smem.xBars[idxXBuf];
        auto& vBar = smem.vBars[idxVBuf];
        // @fixme: merge these two barriers and use test_wait_parity() early to avoid latency.
        vBar.produced.wait_parity(toParity<SharedMemB::nbVBufs>(iter));
        xBar.produced.wait_parity(toParity<SharedMemB::nbXBufs>(iter));
        auto const& xBuf = smem.x(idxXBuf);
        auto const& vBuf = smem.v(idxVBuf)[tileIdx.x];
        auto const xRowSum = loadShmRowMax<warpTile.y>(smem.xRowSum[idxXBuf], tileBase.y, lane);
        accRowSum = accRowSum + xRowSum;
        storeRowMax<warpTile.y>(smem.accRowSum[tileIdx.x], accRowSum, tileBase.y, lane);

#pragma unroll
        for (uint32_t idxInstK = 0; idxInstK < tileNbInstK; idxInstK++)
        {
            Mat16x32Loader const loaderX(xBuf, tileBase.y, idxInstK, rA, cA);
            Vec<Mat16x32, exactDiv(warpTile.y, qmmaShape.m)> const x = loaderX.loadWholeCol<warpTile.y>();
            using AtomB = Vec<uint32_t, 2>;
#pragma unroll
            for (uint32_t idxAtomBx2 = 0; idxAtomBx2 < warpTileNbAtomBx2; idxAtomBx2++)
            {
                auto const data
                    = ldmatrix_16x16_trans<2>(&vBuf.template at<true>(qmmaShape.k * idxInstK + rB, idxAtomBx2 + cB));
                AtomB const v[2] = {data[0], data[2], data[1], data[3]};
                if (idxInstK == tileNbInstK - 1 && idxAtomBx2 == warpTileNbAtomBx2 - 2)
                {
                    uint32_t const iterNext = iter + 1;
                    skipWait_xRowMaxLog2eProducedBar
                        = smem.xRowMaxLog2eProducedBar[iterNext % nbProducerCtasPerCga].test_wait_parity(
                            toParity<nbProducerCtasPerCga>(iterNext));
                }
#pragma unroll
                for (uint32_t i = 0; i < WarpAcc::rows; i++)
                {
#pragma unroll
                    for (uint32_t j = 0; j < 2; j++)
                    {
#if 1
                        mma<__hip_fp8_e4m3_fnuz>(
#else
                        mmaF8_k32_2inst(
#endif
                            reinterpret_cast<float(&)[2][2]>(acc(i, 2 * idxAtomBx2 + j)),
                            reinterpret_cast<uint32_t const(&)[2][2]>(x[i]),
                            reinterpret_cast<uint32_t const(&)[2][1]>(v[j]));
                    }
                }
            }
        }
        bool const isLastIter = (iterToTile(iter + 1) >= nbTiles());
        if (isLastIter)
        {
            idxXVBufLast = idxXBuf;
            assert(idxXBuf == idxVBuf);
        }
        else
        {
            xBar.consumed.arrive();
            vBar.consumed.arrive();
        }
    }

    smem.mathWarpsBar.arrive();

    ThrdRegRowMax const accRowSum = loadShmRowMax<warpTile.y>(smem.accRowSum[tileIdx.x], tileBase.y, lane);
    float const xvScale = computeRowSumFromF8 ? args.kvCacheScale[0] : args.kvCacheScale[0] * xScale;
    WarpOutputTile const output = finalize(acc, accRowSum, xvScale, lane);

    bool const isMultiBlockMode = (nbSubSeq != 1);
    static_assert(PartialResult::nbRowsPerChunk == warpTile.y);
    auto& dst = isMultiBlockMode
        ? args.partialResults[maxNbSubSeq * idxInputTokenGlobal + idxSubSeq].chunks[tileIdx.y].data
        : reinterpret_cast<Vec<OutputHead, warpTile.y>&>(args.output[headGrpSize * idxInputTokenGlobal + tileBase.y]);

    assert(warpRank < nbMathWarps);
    WarpOutSwizzleBuf& swizzleBuf
        = reinterpret_cast<Vec<WarpOutSwizzleBuf, nbWarpOutSwizzleBuf>&>(smem.xv[idxXVBufLast])[warpRank];
    // make sure all math warps have finished using XVBuffer.
    smem.mathWarpsBar.wait_parity(false);

    storeOutput(dst, gemm1V * idxConsumer() + tileBase.x, output, swizzleBuf, lane);
    if (isMultiBlockMode && tileIdx.x == 0)
    {
        ThrdRegRowMax const accRowMaxLog2e
            = loadShmRowMax<warpTile.y>(smem.accRowMaxLog2e[tileIdx.x], tileBase.y, lane);
        auto& chunk = args.partialResults[maxNbSubSeq * idxInputTokenGlobal + idxSubSeq].chunks[tileIdx.y];
#pragma unroll
        for (uint32_t i = 0; i < ThrdRegRowMax::size; i++)
        {
            chunk.rowMaxLog2e[warp_size * i + lane] = accRowMaxLog2e[i];
            chunk.rowSum[warp_size * i + lane] = accRowSum[i];
        }
    }
    smem.xBars[idxXVBufLast].consumed.arrive();
    smem.vBars[idxXVBufLast].consumed.arrive();
}

__device__ inline void Consumer::loadX()
{
#pragma unroll 1
    for (uint32_t iter = 0; true; iter++)
    {
        uint32_t const idxTile = iterToTile(iter);
        if (idxTile >= nbTiles())
        {
            break;
        }
        // @todo: merge these two barriers.
        uint32_t const idxScratchXBuf = iter % nbProducerCtasPerCga;
        auto& srcProducedBar = smem.cgaXBufProduced[idxScratchXBuf];
        srcProducedBar.wait_parity(toParity<nbProducerCtasPerCga>(iter));
        uint32_t const idxXBuf = iter % SharedMemB::nbXBufs;
        auto& xBar = smem.xBars[idxXBuf];
        xBar.consumed.wait_parity(toParity<SharedMemB::nbXBufs>(iter));
        if (warpElectSync())
        {
            auto& src = args.cgaXBuf[nbSubSeq * idxInputTokenGlobal + idxSubSeq][idxScratchXBuf];
            auto& dstX = smem.x(idxXBuf);
            auto& dstRowSum = smem.xRowSum[idxXBuf];
            tma::load1DAsync(&dstX, &src.x, sizeof(smem.x(0)), xBar.produced);
            tma::load1DAsync(&dstRowSum, &src.rowSum, sizeof(smem.xRowSum[0]), xBar.produced);
            xBar.produced.arrive_tx(sizeof(smem.x(0)) + sizeof(smem.xRowSum[0]));
            xBar.produced.wait_parity(toParity<SharedMemB::nbXBufs>(iter));
            uint32_t const idxProducer = idxScratchXBuf;
            // @fixme: check if this works. If it doesn't, randomly pick some data from dstX and dstRowSum and use
            // STAS + arrive_tx to avoid fence.
            getProducerShm(idxProducer).cgaXBufConsumed.arrive<Scope::CGA, ArriveOrder::RELAXED>();
        }
    }
}

__device__ inline void Consumer::loadV()
{
    KVTilePartLoader loader(args.cacheList, idxReq, args.tensorMapV
#if USE_PAGED_KV_CACHE
        ,
        divUp(seqLen, tokensPerPage)
#endif
    );
    for (uint32_t iter = 0; true; iter++)
    {
        uint32_t const idxTile = iterToTile(iter);
        if (idxTile >= nbTiles())
        {
            break;
        }
        uint32_t const idxPageBuf = iter % KVTilePartLoader::nbPageBuffers;
        loader.loadPages(idxTile, idxPageBuf);
        uint32_t const idxVBuf = iter % SharedMemB::nbVBufs;
        auto& vBar = smem.vBars[idxVBuf];
        vBar.consumed.wait_parity(toParity<SharedMemB::nbVBufs>(iter));
#pragma unroll
        for (uint32_t idxPart = 0; idxPart < SharedMemB::VBuffer::size; idxPart++)
        {
            loader.loadData(smem.v(idxVBuf)[idxPart], idxTile,
                gemm1V * idxConsumer() + exactDiv(gemm1V, SharedMemB::VBuffer::size) * idxPart, vBar.produced,
                idxPageBuf);
        }
        if (warpElectSync())
        {
            vBar.produced.arrive_tx(sizeof(SharedMemB::VBuffer));
        }
    }
}

__device__ inline Array2D<uint32_t, InstAcc::rows * Consumer::WarpAcc::rows, Consumer::WarpAcc::cols>
Consumer::finalize(WarpAcc const& acc, ThrdRegRowMax const& accRowSum, float const xvScale, uint32_t const lane)
{
    ThrdRegRowMax const scaleVec = 1.F / (accRowSum) *xvScale;
    WarpOutputTile ret;
#pragma unroll
    for (uint32_t m = 0; m < WarpAcc::rows; m++)
    {
#pragma unroll
        for (uint32_t i = 0; i < InstAcc::rows; i++)
        {
            uint32_t retRow = m * InstAcc::rows + i;
            float const scale = __shfl_sync(~0U, scaleVec[m / 2], 16 * (m % 2) + 8 * i + lane / 4);
#pragma unroll
            for (uint32_t n = 0; n < WarpAcc::cols; n++)
            {
                float data[InstAcc::cols];
#pragma unroll
                for (uint32_t j = 0; j < InstAcc::cols; j++)
                {
                    data[j] = acc(m, n)(i, j) * scale;
                }
                assert(InstAcc::cols == 2);
                reinterpret_cast<__hip_bfloat162&>(ret(retRow, n)) = __float22bfloat162_rn(float2{data[0], data[1]});
            }
        }
    }
    return ret;
}

__device__ inline void Consumer::storeOutput(Vec<OutputHead, warpTile.y>& dst, uint32_t dstBaseCol,
    WarpOutputTile const& src, WarpOutSwizzleBuf& swizzleBuf, uint32_t lane)
{
    using Dst = mha::decay_t<decltype(dst)>;
    static_assert(Dst::size == WarpOutputTile::rows * 8 && Dst::size % WarpOutSwizzleBuf::rows == 0);
    uint32_t const nbIters = exactDiv(Dst::size, WarpOutSwizzleBuf::rows);

    uint32_t const rS = lane % 8;
    uint32_t const cS = lane / 8;

    uint32_t const thrdsPerRow = exactDiv(sizeof(WarpOutSwizzleBuf::Elem) * WarpOutSwizzleBuf::cols, grainBytes);
    static_assert(thrdsPerRow <= 32);
    uint32_t const rL = lane / thrdsPerRow;
    uint32_t const cL = lane % thrdsPerRow;
#pragma unroll
    for (uint32_t iter = 0; iter < nbIters; iter++)
    {
#pragma unroll
        for (uint32_t j = 0; j < WarpOutputTile::cols; j += 4)
        {
            auto const baseSwzPtr = &swizzleBuf.template at<true>(rS, j + cS);
            constexpr uint32_t srcRowsPerIter = exactDiv(WarpOutputTile::rows, nbIters);
#pragma unroll
            for (uint32_t i = 0; i < srcRowsPerIter; i++)
            {
                static_assert(sizeof(WarpOutSwizzleBuf::Elem) * WarpOutSwizzleBuf::cols * 8 % 1024 == 0);
                auto const swzPtr = checkedVal(
                    baseSwzPtr + WarpOutputTile::cols * 8 * i, &swizzleBuf.template at<true>(8 * i + rS, j + cS));
                stmatrix<false, 4>(
                    swzPtr, reinterpret_cast<Vec<uint32_t, 4> const&>(src(srcRowsPerIter * iter + i, j)));
            }
        }
        __syncwarp();

        uint32_t const dstRowsPerIter = WarpOutSwizzleBuf::rows;
        uint32_t const rowsPerOp = exactDiv(warp_size, thrdsPerRow);
        LdGrain* const baseDstPtr = reinterpret_cast<LdGrain*>(
            &dst[dstRowsPerIter * iter + rL][dstBaseCol + exactDiv(grainBytes, sizeof(OutputElem)) * cL]);
#pragma unroll
        for (uint32_t i = 0; i < dstRowsPerIter; i += rowsPerOp)
        {
            LdGrain* const dstPtr = checkedVal(baseDstPtr + i * exactDiv(sizeof(OutputHead), grainBytes),
                reinterpret_cast<LdGrain*>(
                    &dst[dstRowsPerIter * iter + i + rL][dstBaseCol + exactDiv(grainBytes, sizeof(OutputElem)) * cL]));
            LdGrain* const srcPtr = &swizzleBuf.template at<true>(i + rL, cL);
            *dstPtr = *srcPtr;
        }
        __syncwarp();
    }
}

__device__ inline void mergePartialOutputs(uint32_t& semaphore, Vec<OutputHead, PartialResult::nbRowsPerChunk>& dst,
    PartialResult const* reqPartialResults, uint32_t nbSubSeq, uint32_t ctaRank, uint32_t warpRank, uint2 warpIdx,
    void* sharedMem)
{
    assert(nbSubSeq > 1);
    clusterBarArrive();
    clusterBarWait();
    bool const isProducer = (ctaRank < nbProducerCtasPerCga);

    bool& shmIsLastSubSeq = isProducer ? static_cast<SharedMemA*>(sharedMem)->isLastSubSeq
                                       : static_cast<SharedMemB*>(sharedMem)->isLastSubSeq;

    if (ctaRank == 3 && threadIdx.x == 0)
    {
        uint32_t old;
        uint32_t const lastOld = nbSubSeq - 1;
        asm volatile("atom.relaxed.gpu.global.inc.u32 %0, [%1], %2;\n" : "=r"(old) : "l"(&semaphore), "r"(lastOld));
        bool const isLastSubSeq = (old == lastOld);
#pragma unroll
        for (uint32_t i = 0; i < nbProducerCtasPerCga; i++)
        {
            static_cast<SharedMemA*>(mapa(sharedMem, i))->isLastSubSeq = isLastSubSeq;
        }
        mapa(shmIsLastSubSeq, 2) = isLastSubSeq;
        shmIsLastSubSeq = isLastSubSeq;
    }
    clusterBarArrive();
    clusterBarWait();
    bool const isLastCga = shmIsLastSubSeq;
    if (!isLastCga)
    {
        return;
    }

    CtaBarrierPair(&bars)[nbMultiBlockBufs] = isProducer ? static_cast<SharedMemA*>(sharedMem)->multiBlockBars
                                                         : static_cast<SharedMemB*>(sharedMem)->multiBlockBars;
    Vec<PartialResult::Chunk, nbMultiBlockBufs>& shmBufs = isProducer
        ? static_cast<SharedMemA*>(sharedMem)->getMultiBlockBufs()
        : static_cast<SharedMemB*>(sharedMem)->getMultiBlockBufs();

    constexpr uint32_t nbShmBufs = nbMultiBlockBufs;

    if (warpIdx.y == 2)
    {
        asm volatile("setmaxnreg.dec.sync.aligned.u32 %0;\n" ::"n"(nbRegsForIOWarps));
        if (warpIdx.x == 0)
        {
#pragma unroll 1
            for (uint32_t idxSubSeq = 0; idxSubSeq < nbSubSeq; idxSubSeq++)
            {
                uint32_t const idxBuf = idxSubSeq % nbShmBufs;
                auto& bar = bars[idxBuf];
                bar.consumed.wait_parity(toParity<nbShmBufs>(idxSubSeq));
                if (warpElectSync())
                {
                    tma::load1DAsync(&shmBufs[idxBuf], &reqPartialResults[idxSubSeq].chunks[ctaRank],
                        sizeof(PartialResult::Chunk), bar.produced);
                    bar.produced.arrive_tx(sizeof(PartialResult::Chunk));
                }
            }
        }
    }
    else
    {
        asm volatile("setmaxnreg.inc.sync.aligned.u32 %0;\n" ::"n"(nbRegsForMathWarps));
        constexpr uint32_t nbMathWarps = 8;
        constexpr uint32_t rowsPerWarp = exactDiv(PartialResult::nbRowsPerChunk, nbMathWarps);
        constexpr uint32_t regGrainsPerRow = exactDiv(sizeof(OutputHead), grainBytes * warp_size);
        constexpr uint32_t grainOutElems = exactDiv(grainBytes, sizeof(OutputElem));
        uint32_t const lane = laneId();

        uint32_t const tileRowBase = rowsPerWarp * warpRank;
        using RowWise = Vec<float, rowsPerWarp>;
        using RegChunk = Array2D<Vec<OutputElem, grainOutElems>, rowsPerWarp, regGrainsPerRow>;
        auto loadBuf = [&](RowWise& rowMaxLog2e, RowWise& rowSum, RegChunk& regChunk, PartialResult::Chunk const& chunk)
        {
            auto loadRowWise = [&](Vec<float, PartialResult::nbRowsPerChunk> const& src)
            { return reinterpret_cast<RowWise const&>(src[tileRowBase]); };
            rowMaxLog2e = loadRowWise(chunk.rowMaxLog2e);
            rowSum = loadRowWise(chunk.rowSum);
            regChunk;
#pragma unroll
            for (uint32_t i = 0; i < rowsPerWarp; i++)
            {
#pragma unroll
                for (uint32_t j = 0; j < regGrainsPerRow; j++)
                {
                    regChunk(i, j) = reinterpret_cast<Vec<OutputElem, grainOutElems> const&>(
                        chunk.data[tileRowBase + i][grainOutElems * (warp_size * j + lane)]);
                }
            }
        };

        uint32_t const idxSubSeqInit = 0;
        uint32_t const idxBufInit = idxSubSeqInit % nbShmBufs;
        bars[idxBufInit].produced.wait_parity(toParity<nbShmBufs>(idxSubSeqInit));
        RowWise accRowMaxLog2e;
        RowWise accRowSum;
        RegChunk chunk;
        loadBuf(accRowMaxLog2e, accRowSum, chunk, shmBufs[idxBufInit]);
        bars[idxBufInit].consumed.arrive();

        using Acc = Array2D<Vec<float, grainOutElems>, rowsPerWarp, regGrainsPerRow>;
        Acc acc;
#pragma unroll
        for (uint32_t i = 0; i < rowsPerWarp; i++)
        {
#pragma unroll
            for (uint32_t j = 0; j < regGrainsPerRow; j++)
            {
                acc(i, j) = convert<float>(chunk(i, j)) * accRowSum[i];
            }
        }

#pragma unroll 1
        for (uint32_t idxSubSeq = idxSubSeqInit + 1; idxSubSeq < nbSubSeq; idxSubSeq++)
        {
            uint32_t const idxBuf = idxSubSeq % nbShmBufs;
            auto& bar = bars[idxBuf];
            bar.produced.wait_parity(toParity<nbShmBufs>(idxSubSeq));
            RowWise chunkRowMaxLog2e;
            RowWise chunkRowSum;
            loadBuf(chunkRowMaxLog2e, chunkRowSum, chunk, shmBufs[idxBuf]);
            bar.consumed.arrive();
#pragma unroll
            for (uint32_t i = 0; i < rowsPerWarp; i++)
            {
                bool const newChunkGreater = (chunkRowMaxLog2e[i] > accRowMaxLog2e[i]);
                if (newChunkGreater)
                {
                    float const scale = exp2f(accRowMaxLog2e[i] - chunkRowMaxLog2e[i]);
#pragma unroll
                    for (uint32_t j = 0; j < regGrainsPerRow; j++)
                    {
                        acc(i, j) = acc(i, j) * scale + convert<float>(chunk(i, j)) * chunkRowSum[i];
                    }
                    accRowSum[i] = accRowSum[i] * scale + chunkRowSum[i];
                    accRowMaxLog2e[i] = chunkRowMaxLog2e[i];
                }
                else
                {
                    float const scale = exp2f(chunkRowMaxLog2e[i] - accRowMaxLog2e[i]);
                    float const fusedScale = scale * chunkRowSum[i];
#pragma unroll
                    for (uint32_t j = 0; j < regGrainsPerRow; j++)
                    {
                        acc(i, j) = acc(i, j) + convert<float>(chunk(i, j)) * fusedScale;
                    }
                    accRowSum[i] = accRowSum[i] + chunkRowSum[i] * scale;
                }
            }
        }

#pragma unroll
        for (uint32_t i = 0; i < rowsPerWarp; i++)
        {
            float const scale = 1.F / accRowSum[i];
            auto const dstHead = reinterpret_cast<Vec<OutputElem, grainOutElems>*>(&dst[tileRowBase + i]);
#pragma unroll
            for (uint32_t j = 0; j < regGrainsPerRow; j++)
            {
                dstHead[warp_size * j + lane] = convert<OutputElem>(acc(i, j) * scale);
            }
        }
    }
}

inline constexpr uint32_t cgaSize = nbProducerCtasPerCga + nbVSplit;

CUBIN_EXPORT __global__ __launch_bounds__(32 * 4 * 3, 1) __cluster_dims__(cgaSize, 1, 1) void kernel_mha(
    __grid_constant__ CUtensorMap const tensorMapQ, // MhaIOHead[nbQHeads * totalNbInputTokens],
    __grid_constant__ CUtensorMap const tensorMapK, // with box=64 for the least significant dim
    __grid_constant__ CUtensorMap const tensorMapV, // with box=128 for the least significant dim
    float const qScale,
    OutputHead* __restrict__ const output,          // [totalNbIntputTokens][nbQHeads]
    KVCacheList<usePagedKVCache> const cacheList, uint32_t const batchSize,
    float const* __restrict__ const kvCacheScale,   // Device memory scalar. Same scale for K and V cache. Used only for
                                                    // int8/fp8 KV cache.
    Vec<CgaXBuffer, nbProducerCtasPerCga>* __restrict__ const cgaXBuf, // [totalNbInputTokens][maxNbSubSeq]
    uint32_t* __restrict__ const semaphores = nullptr,                 // [totalNbInputTokens]
    PartialResult* __restrict__ const partialResults = nullptr)        // [totalNbInputTokens][maxNbSubSeq]
{
    assert(blockDim.x == 32 * 12 && blockDim.y == 1 && blockDim.z == 1);
    extern __shared__ char smemBuf[];
    uint32_t const warpRank = makeWarpUniform(this_warp(), threadIdx.x / warp_size);
    uint2 const warpIdx = {warpRank % 4, warpRank / 4};

    uint3 const& cgaId = clusterId();
    uint32_t const& idxReq = cgaId.z;
    uint32_t const& maxNbSubSeq = nbClusters().y;
    uint32_t const& idxSubSeq = cgaId.y;
    uint32_t const inputSeqLen
        = (allowMultipleInputTokens ? exactDiv(gridDim.x, cgaSize) : checkedVal(1U, exactDiv(gridDim.x, cgaSize)));
    uint32_t const reqIdxInputToken
        = (allowMultipleInputTokens ? blockIdx.x / cgaSize : checkedVal(0U, blockIdx.x / cgaSize));
    uint32_t const idxInputTokenGlobal = inputSeqLen * idxReq + reqIdxInputToken;
    uint32_t const cacheSeqLen = cacheList.seqLenList[idxReq] - (inputSeqLen - 1) + reqIdxInputToken;
    assert(beamWidth == 1);
    uint32_t const nbTiles = useKVCache ? divUp(cacheSeqLen, tokensPerTile) : 0;
    bool const isMultiBlockMode = (maxNbSubSeq > 1 && nbTiles >= multiBlockMinNbTiles);
    uint32_t const nbSubSeq = isMultiBlockMode ? mha::min(nbTiles / multiBlockMinNbTilesPerCta, maxNbSubSeq) : 1;
    static_assert(multiBlockMinNbTiles >= multiBlockMinNbTilesPerCta * 2);
    assert(isMultiBlockMode == (nbSubSeq > 1));
    if (idxSubSeq >= nbSubSeq)
    {
        return;
    }

    uint32_t const ctaRank = clusterCtaRank();
    bool const isProducer = (ctaRank < nbProducerCtasPerCga);

    KernelArgs const args{tensorMapQ, tensorMapK, tensorMapV, qScale, output, cacheList, batchSize, kvCacheScale,
        cgaXBuf, semaphores, partialResults};

    if (isProducer)
    {
        Producer{args, *reinterpret_cast<SharedMemA*>(smemBuf), maxNbSubSeq, idxReq, idxInputTokenGlobal, cacheSeqLen,
            nbSubSeq, idxSubSeq, ctaRank, warpRank, warpIdx}
            .run();
    }
    else
    {
        Consumer{args, *reinterpret_cast<SharedMemB*>(smemBuf), maxNbSubSeq, idxReq, idxInputTokenGlobal, cacheSeqLen,
            nbSubSeq, idxSubSeq, ctaRank, warpRank, warpIdx}
            .run();
    }
}

__constant__ constexpr uint32_t smemSize = mha::max(sizeof(SharedMemA), sizeof(SharedMemB));
static_assert(smemSize <= 99 * 1024, "Shared memory size exceeded");
#endif // is_MLA

#ifndef GENERATE_CUBIN
#if IS_MLA
CUtensorMap makeTensorMapForQ(
    void const* addr, CUtensorMapDataType_enum dataType, uint32_t headElems, uint32_t totalNbHeads, uint32_t partElems)
{
    CUtensorMap tensorMap{};
    uint64_t const globalDims[] = {headElems, totalNbHeads};
    uint32_t elemBytes = getElemBytes(dataType);
    uint32_t const headBytes = elemBytes * headElems;
    uint64_t const globalStrides[] = {headBytes};
    uint32_t const boxDims[] = {partElems, headGrpSize};
    uint32_t const elemStrides[] = {1, 1};
    auto const swizzle = CU_TENSOR_MAP_SWIZZLE_64B;

    checkCu(cuTensorMapEncodeTiled(&tensorMap, dataType, 2, const_cast<void*>(addr), globalDims, globalStrides, boxDims,
        elemStrides, CU_TENSOR_MAP_INTERLEAVE_NONE, swizzle, CU_TENSOR_MAP_L2_PROMOTION_NONE,
        CU_TENSOR_MAP_FLOAT_OOB_FILL_NONE));
    return tensorMap;
}
#endif // IS_MLA

void launchMLA(hipDeviceProp_t const& prop,
    uint32_t inputSeqLen, // uniform for all requests and causal mask is assumed
    float qScale, OutputHead* output, InputHead const* q,
#if USE_PAGED_KV_CACHE
    GMemCacheHead* pool, // global pool of pages
    KVCachePageIndex const*
        kvCachePageList, // device pointer. shape: KVCachePage[batchSize][beamWidth][2][maxNbPagesPerSeq]
#else
    GMemKVCacheHead* kvCacheData,
#endif
    uint32_t maxSeqLen, uint32_t const* seqLen, uint32_t batchSize,
    float const* __restrict__ kvCacheScale, // Device memory scalar. Same scale for K and V cache. Used only for
                                            // int8/fp8 KV cache.
    uint32_t* semaphores, void* scratch, hipStream_t stream)
{
#if IS_MLA
    static_assert(
        SLIDING_WINDOW == 0 && LOW_PREC_OUTPUT == 0 && USE_INPUT_KV == 0 && USE_BEAM_SEARCH == 0, "not implemented");
    if (beamWidth != 1)
    {
        throw std::runtime_error("not implemented");
    }
    static uint32_t const hostSmemSize = [&]()
    {
        // printf("smemSize = %u\n", smemSize);
        uint32_t size;
        checkCuda(hipMemcpyFromSymbol(&size, HIP_SYMBOL(smemSize), sizeof(smemSize)));
        checkCuda(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_mha), hipFuncAttributeMaxDynamicSharedMemorySize, size));
        return size;
    }();
    uint32_t const nbKHeads = 1;
    uint32_t const nbVHeads = nbKHeads;
    uint32_t const nbQHeads = nbKHeads * headGrpSize;
    uint32_t const nbQKVHeads = nbQHeads + nbKHeads + nbVHeads;
    uint32_t const nbSubSeqPerSeq = [&]() -> uint32_t
    {
        auto const env = std::getenv("XQA_NB_SUB_SEQ");
        if (env != nullptr)
        {
            int32_t const val = std::stoi(env);
            if (val > 0)
            {
                return val;
            }
        }
        float const factor = 4.f;
        return mha::min<uint32_t>(
            mha::max<uint32_t>(1U, (uint32_t) round(prop.multiProcessorCount / 4 / (batchSize * nbKHeads) * factor)),
            divUp(maxSeqLen, tokensPerTile * 2));
    }();
    // printf("nbSubSeqPerSeq = %u\n", nbSubSeqPerSeq);
    // gridDim.z == nbKHeads * batchSize && gridDim.y == nbSubSeqPerSeq && gridDim.x == nbInputSeqSplit
    dim3 const dimGrid{4 * inputSeqLen, nbSubSeqPerSeq, nbKHeads * batchSize};
    dim3 const dimCta{warp_size * 4 * 3, 1, 1};
    auto const launchCfg = makeLaunchConfig(dimGrid, dimCta, hostSmemSize, stream, ENABLE_PDL != 0);
#if USE_PAGED_KV_CACHE
    uint32_t const maxNbPagesPerSeq = exactDiv(maxSeqLen, tokensPerPage);
    KVCacheList<true> const cacheList{pool, kvCachePageList, seqLen, maxNbPagesPerSeq};
    auto const dtype = []
    {
        if (std::is_same_v<CacheElem, half>)
        {
            return CU_TENSOR_MAP_DATA_TYPE_FLOAT16;
        }
        else if (std::is_same_v<CacheElem, __hip_bfloat16>)
        {
            return CU_TENSOR_MAP_DATA_TYPE_BFLOAT16;
        }
        else if (std::is_same_v<CacheElem, __hip_fp8_e4m3_fnuz>)
        {
            return CU_TENSOR_MAP_DATA_TYPE_UINT8;
        }
        throw std::runtime_error("unsupported cache element type");
    }();

    auto const tensorMapQ
        = makeTensorMapForQ(q, dtype, validElemsPerHead, headGrpSize * inputSeqLen * batchSize, partElemsK);
    auto const tensorMapK = makeTensorMapForPagedKVCache(
        pool, dtype, validElemsPerHead, nbKHeads, tokensPerPage, partElemsK, tokensPerTile);
    auto const tensorMapV = makeTensorMapForPagedKVCache(
        pool, dtype, validElemsPerHead, nbKHeads, tokensPerPage, partElemsV, tokensPerTile);

    uint32_t const nbCgas = exactDiv(dimGrid.x, 4) * dimGrid.y * dimGrid.z;
    auto const cgaXBuf = static_cast<Vec<CgaXBuffer, nbProducerCtasPerCga>*>(scratch);
    auto const partialResults = reinterpret_cast<PartialResult*>(cgaXBuf + nbCgas);
    hipError_t const err = cudaLaunchKernelEx(&launchCfg, &kernel_mha, tensorMapQ, tensorMapK, tensorMapV, qScale,
        output, cacheList, batchSize, kvCacheScale, cgaXBuf, semaphores, partialResults);
#else
    KVCacheList<false> const cacheList{kvCacheData, seqLen, maxSeqLen};
    static_assert(!usePagedKVCache);
    assert(gemm0CtaTileNbTokens == gemm1CtaTileNbTokens);
    auto const tensorMap = makeTensorMapForContiguousKVCache(kvCacheData, CU_TENSOR_MAP_DATA_TYPE_UINT8,
        validElemsPerHead, nbKHeads, maxSeqLen, beamWidth, batchSize, gemm0CtaTileNbTokens);
    cudaLaunchKernelEx(&launchCfg, kernel_mha, nbKHeads,
#if SLIDING_WINDOW
        slidingWinSize,
#endif
        qScale, output,
#if LOW_PREC_OUTPUT
        rcpOutScale,
#endif
#if USE_INPUT_KV
        qkv,
#if ROPE_STYLE != 0
        ropeCosSin,
#endif
#else
        q,
#endif
        cacheList,
#if USE_BEAM_SEARCH
        beamSearchParams,
#endif
        batchSize, kvCacheScale, tensorMap, semaphores, scratch);
#endif
    checkCuda(err);
#endif
}
#endif
