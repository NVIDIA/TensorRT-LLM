#include "hip/hip_runtime.h"
#include "../utils.h"
#include <cstdint>
#include <hip/hip_runtime.h>

__global__ void kernel_warmup(uint64_t cycles)
{
    uint64_t const tic = clock64();
    while (tic + cycles < clock64())
    {
    }
}

void warmup(hipDeviceProp_t const& prop, float ms, hipStream_t stream = nullptr)
{
    uint64_t const nbCycles = std::round(prop.clockRate * ms); // clockRate is in kHz
    kernel_warmup<<<16, 128, 0, stream>>>(nbCycles);
    checkCuda(hipGetLastError());
}
