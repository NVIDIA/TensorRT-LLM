#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2011-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: NVIDIA TensorRT Source Code License Agreement
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "fused_multihead_attention_dgrad_kernel_1xN_reload.h"
#include "fused_multihead_attention_dgrad_kernel_1xN_reload_noloop.h"
#include "fused_multihead_attention_fprop.h"
// #include "fused_multihead_attention_dgrad_kernel_1xN_noloop.h"
//

using Kernel_traits = fmha::Kernel_traits_v2<fmha::Ampere_hmma_fp32_traits, 512, 64, 16, 1, 8, 1, 0x28u>;
static_assert(!Kernel_traits::HEADS_INTERLEAVED);

extern "C" __global__ void fmha_dgrad_v2_fp16_512_64_sm80_kernel(Fused_multihead_attention_fprop_params params)
{
    fused_multihead_attention::compute_dv_1xN<Kernel_traits>(params);
    fused_multihead_attention::compute_dq_dk_1xN<Kernel_traits>(params);
}

template <int CHUNKS>
__global__ void fmha_dgrad_v2_fp16_512_64_sm80_noloop_kernel(Fused_multihead_attention_fprop_params params)
{
    // TODO make CHUNKS parameter runtime in the kernels
    fused_multihead_attention::compute_dv_1xN_nl<CHUNKS, Kernel_traits>(params);
    fused_multihead_attention::compute_dq_dk_1xN_nl<CHUNKS, Kernel_traits>(params);
}

void run_fmha_dgrad_v2_fp16_512_64_sm80(Fused_multihead_attention_fprop_params const& params, hipStream_t stream)
{

    constexpr int smem_size_softmax = Kernel_traits::Cta_tile_p::M * Kernel_traits::Cta_tile_p::WARPS_N * sizeof(float);
    constexpr int smem_size_q = Kernel_traits::Smem_tile_q::BYTES_PER_TILE;
    constexpr int smem_size_v = Kernel_traits::Smem_tile_v::BYTES_PER_TILE;
    constexpr int smem_size_o = Kernel_traits::Smem_tile_o::BYTES_PER_TILE;

    using Smem_tile_s = Smem_tile_mma_transposed<fmha::Ampere_hmma_fp32_traits, Kernel_traits::Cta_tile_p>;
    constexpr int smem_size_s = Smem_tile_s::BYTES_PER_TILE;
    static_assert(smem_size_s == 16 * 512 * 2);
    static_assert(smem_size_o == 16 * 64 * 4 * Kernel_traits::Cta_tile_p::WARPS_N);

    constexpr int smem_size_dv = smem_size_s + 2 * smem_size_q + smem_size_v + smem_size_softmax;
    constexpr int smem_size_dq_dk = smem_size_s + smem_size_o + smem_size_q + smem_size_v;
    constexpr int smem_size = std::max(smem_size_dv, smem_size_dq_dk);

    if (smem_size >= 48 * 1024)
    {
        FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(
            fmha_dgrad_v2_fp16_512_64_sm80_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
    }

    dim3 grid(params.h, params.b, 1);
    fmha_dgrad_v2_fp16_512_64_sm80_kernel<<<grid, Kernel_traits::THREADS, smem_size, stream>>>(params);
}

void run_fmha_dgrad_v2_fp16_512_64_sm80_noloop(
    Fused_multihead_attention_fprop_params const& params, int const num_chunks, hipStream_t stream)
{

    constexpr int smem_size_softmax = Kernel_traits::Cta_tile_p::M * Kernel_traits::Cta_tile_p::WARPS_N * sizeof(float);
    constexpr int smem_size_q = Kernel_traits::Smem_tile_q::BYTES_PER_TILE;
    constexpr int smem_size_v = Kernel_traits::Smem_tile_v::BYTES_PER_TILE;
    constexpr int smem_size_o = Kernel_traits::Smem_tile_o::BYTES_PER_TILE;

    using Smem_tile_s = Smem_tile_mma_transposed<fmha::Ampere_hmma_fp32_traits, Kernel_traits::Cta_tile_p>;
    constexpr int smem_size_s = Smem_tile_s::BYTES_PER_TILE;
    static_assert(smem_size_s == 16 * 512 * 2);
    static_assert(smem_size_o == 16 * 64 * 4 * Kernel_traits::Cta_tile_p::WARPS_N);

    constexpr int smem_size_dv = smem_size_s + 2 * smem_size_q + smem_size_v + smem_size_softmax;
    constexpr int smem_size_dq_dk = smem_size_s + smem_size_o + smem_size_q + smem_size_v;
    constexpr int smem_size = std::max(smem_size_dv, smem_size_dq_dk);

    auto kernel = fmha_dgrad_v2_fp16_512_64_sm80_noloop_kernel<2>;

    if (num_chunks == 2)
    {
        kernel = fmha_dgrad_v2_fp16_512_64_sm80_noloop_kernel<2>;
    }
    else if (num_chunks == 3)
    {
        kernel = fmha_dgrad_v2_fp16_512_64_sm80_noloop_kernel<3>;
    }
    else if (num_chunks == 4)
    {
        kernel = fmha_dgrad_v2_fp16_512_64_sm80_noloop_kernel<4>;
    }
    else
    {
        assert(false && "Unsupperted number of chunks");
    }

    if (smem_size >= 48 * 1024)
    {
        FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
    }

    dim3 grid(params.h, params.b, num_chunks);

    kernel<<<grid, Kernel_traits::THREADS, smem_size, stream>>>(params);

    FMHA_CHECK_CUDA(hipPeekAtLastError());
}
