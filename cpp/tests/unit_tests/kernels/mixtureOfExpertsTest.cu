#include "hip/hip_runtime.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/cutlass_kernels/cutlass_preprocessors.h"
#include "tensorrt_llm/runtime/cudaStream.h"

#include <algorithm>
#include <gtest/gtest.h>
#include <numeric>

#include "tensorrt_llm/kernels/internal_cutlass_kernels/include/moe_kernels.h"
#include "tensorrt_llm/runtime/bufferManager.h"

#include <tensorrt_llm/kernels/cutlass_kernels/cutlass_type_conversion.h>
#include <tensorrt_llm/kernels/quantization.h>

using namespace tensorrt_llm::kernels;
using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

constexpr static float FP8_MAX = 440.f;
constexpr static float FP4_MAX = 6.f;

template <bool IS_SCALED_TYPE>
__host__ __device__ constexpr float applyExpertShift(float weight_value, int expert, int num_experts)
{
    if (IS_SCALED_TYPE && num_experts < 64)
    {
        // Use a power of two centred on 1
        return weight_value * pow(2.0f, expert - num_experts / 2);
    }
    else
    {
        return weight_value + (float) expert / num_experts;
    }
}

template <class T, bool IS_SCALED_TYPE>
__global__ void initWeightsKernel(T* data, int64_t w, int64_t h, float base, float scale)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
    {
        start_offset[y * w + x]
            = (x == y) ? T(applyExpertShift<IS_SCALED_TYPE>(base * scale, expert_id, gridDim.z)) : T(0.f);
    }
}

template <class T, bool IS_SCALED_TYPE>
__global__ void initWeightsGatedKernel(T* data, int64_t w, int64_t h, float base_1, float base_2, float scale)
{
    size_t expert_id = blockIdx.z;
    T* start_offset = data + expert_id * w * h * 2;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    size_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < w && y < h)
    {
        start_offset[y * w + x]
            = (x == y) ? T(applyExpertShift<IS_SCALED_TYPE>(base_1 * scale, expert_id, gridDim.z)) : T(0.f);
        start_offset[(y + h) * w + x]
            = (x == y) ? T(applyExpertShift<IS_SCALED_TYPE>(base_2 * scale, expert_id, gridDim.z)) : T(0.f);
    }
}

template <class T>
__global__ void initBiasToExpertIdKernel(T* data, int64_t w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
        start_offset[x] = T(expert_id);
}

template <class T>
__global__ void initBiasToExpertIdGatedKernel(T* data, int64_t w)
{
    size_t expert_id = blockIdx.y;
    T* start_offset = data + expert_id * w * 2;

    size_t x = blockIdx.x * blockDim.x + threadIdx.x;
    if (x < w)
    {
        start_offset[x] = T(expert_id);
        start_offset[x + w] = T(expert_id + 1);
    }
}

template <class T>
using sizeof_bits = cutlass::sizeof_bits<typename cutlass_kernels::TllmToCutlassTypeAdapter<std::remove_cv_t<T>>::type>;

#ifdef ENABLE_FP8
using SafeFP8 = __hip_fp8_e4m3_fnuz;
#else
using SafeFP8 = void;
#endif
#ifdef ENABLE_FP4
using SafeFP4 = __hip_fp4_e2m1;
#else
using SafeFP4 = void;
#endif

template <class TypeTuple_>
class MixtureOfExpertsTest : public ::testing::Test
{
protected:
    using GemmDataType = typename TypeTuple_::DataType;
    using WeightType = typename TypeTuple_::WeightType;
    using OutputType = typename TypeTuple_::OutputType;
    constexpr static bool INT4 = std::is_same_v<WeightType, cutlass::uint4b_t>;
    constexpr static bool FP8 = std::is_same_v<GemmDataType, SafeFP8>;
    constexpr static bool FP4 = std::is_same_v<GemmDataType, SafeFP4>;
    constexpr static bool INT_QUANT = !std::is_same_v<GemmDataType, WeightType>;
    constexpr static int WEIGHT_ELEM_PER_BYTE = (INT4 || FP4) ? 2 : 1;
    using WeightStorage = std::conditional_t<WEIGHT_ELEM_PER_BYTE == 2, uint8_t, WeightType>;
    constexpr static int64_t HIDDEN_SIZE_MULTIPLIER = 16;
    constexpr static int64_t MINIMUM_BYTE_ALIGNMENT = 64;
    constexpr static int64_t MINIMUM_ALIGNMENT = MINIMUM_BYTE_ALIGNMENT * 8 / sizeof_bits<WeightType>::value;
    constexpr static int64_t DEFAULT_HIDDEN_SIZE = HIDDEN_SIZE_MULTIPLIER * MINIMUM_ALIGNMENT;
    constexpr static bool IS_SCALED_TYPE = true;

    // FP4 uses the unquantized data type for inputs and quantizes on the fly
    using DataType = std::conditional_t<FP4, OutputType, GemmDataType>;

    static BufferManager::CudaStreamPtr mStream;
    static std::unique_ptr<BufferManager> mBufferManager;
    static int mDeviceCount;

    std::vector<BufferManager::IBufferPtr> managed_buffers;
    float* mInputProbabilities{};
    DataType* mInputTensor{};

    int64_t mHiddenSize{};
    int64_t mNumExperts{};
    int64_t mK{};

    float getTolerance(float scale = 1.f)
    {
        bool loose_fp8 = mActType != tensorrt_llm::ActivationType::Relu || moeRoutingNeedsRenorm(mNormMode);
        float tol = std::is_same_v<WeightType, uint8_t>     ? 0.1
            : std::is_same_v<WeightType, cutlass::uint4b_t> ? 0.1
            : std::is_same_v<GemmDataType, float>           ? 0.001
            : std::is_same_v<GemmDataType, half>            ? 0.005
            : std::is_same_v<GemmDataType, __hip_bfloat16>   ? 0.05
            : std::is_same_v<GemmDataType, SafeFP8>         ? (loose_fp8 ? 0.06 : 0.001)
            : std::is_same_v<GemmDataType, SafeFP4>         ? 0.05
                                                            : 0.0;

        // Keep the scale in a sane range
        scale = std::min(scale, 30.f);
        return std::max(tol, scale * tol);
    }

    static bool shouldSkip()
    {
#ifndef ENABLE_FP8
        static_assert(!FP8, "FP8 Tests enabled on unsupported CUDA version");
#endif
        bool should_skip_no_device = mDeviceCount <= 0;
        bool should_skip_unsupported_fp8 = getSMVersion() < 89 && FP8;
        bool should_skip_unsupported_fp4 = getSMVersion() < 100 && FP4;
        return should_skip_no_device || should_skip_unsupported_fp8 || should_skip_unsupported_fp4;
    }

    static void SetUpTestCase()
    {
        mDeviceCount = getDeviceCount();
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping due to no/unsupported GPU";
        }

        mStream = std::make_shared<CudaStream>();
        mBufferManager = std::make_unique<BufferManager>(mStream);
    }

    static void TearDownTestCase()
    {
        mBufferManager.reset();
        mStream.reset();
    }

    void SetUp() override
    {
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping due to no/unsupported GPU";
        }
        assert(mBufferManager);
    }

    void TearDown() override
    {
        managed_buffers.clear();
    }

    void initWeights(DataType* buffer, int64_t w, int64_t h, float base, float scalar)
    {
        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsKernel<DataType, IS_SCALED_TYPE><<<grid, block, 0, mStream->get()>>>(buffer, w, h, base, scalar);
    }

    void initBias(DataType* buffer, int64_t w)
    {
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    void initWeightsGated(DataType* buffer, int64_t w, int64_t h, float base_1, float base_2, float scalar)
    {
        if (!mIsGated)
            return initWeights(buffer, w, h, base_1, scalar);

        h /= 2;
        dim3 block(16, 16, 1);
        dim3 grid(divUp(w, block.x), divUp(h, block.y), mNumExperts);
        initWeightsGatedKernel<DataType, IS_SCALED_TYPE>
            <<<grid, block, 0, mStream->get()>>>(buffer, w, h, base_1, base_2, scalar);
    }

    void initBiasGated(DataType* buffer, int64_t w)
    {
        if (!mIsGated)
            return initBias(buffer, w);

        w /= 2;
        dim3 block(256, 1, 1);
        dim3 grid(divUp(w, block.x), mNumExperts);
        initBiasToExpertIdGatedKernel<DataType><<<grid, block, 0, mStream->get()>>>(buffer, w);
    }

    CutlassMoeFCRunner<GemmDataType, WeightType, OutputType> mMoERunner{};
    char* mWorkspace{};
    float* mScaleProbs{};
    DataType* mRawExpertWeight1{};
    DataType* mRawExpertWeight2{};
    WeightStorage* mExpertWeight1{};
    WeightStorage* mExpertWeight2{};
    DataType* mExpertIntScale1{};
    DataType* mExpertIntScale2{};

    float mFP8WeightScalar1{1.f};
    float mFP8WeightScalar2{1.f};
    float* mExpertFPXScale1{};
    float* mExpertFPXScale2{};
    float* mExpertFPXScale3{};

    float* mExpertFP4ActGlobalScale1{};
    float* mExpertFP4WeightGlobalScale1{};
    float* mExpertFP4WeightGlobalScale2{};

    using TmaWarpSpecializedGroupedGemmInput = tensorrt_llm::TmaWarpSpecializedGroupedGemmInput;
    using ElementSF = TmaWarpSpecializedGroupedGemmInput::ElementSF;
    constexpr static int FP4VecSize = TmaWarpSpecializedGroupedGemmInput::BlockScaleVectorSize;
    ElementSF* mFP4ScalingFactorsW1 = nullptr;
    ElementSF* mFP4ScalingFactorsW2 = nullptr;

    DataType* mExpertBias1{};
    DataType* mExpertBias2{};

    void* mTpExpertScratch{}; // Copy the experts here when slicing up inputs
    size_t mTpExpertScratchSize{};

    OutputType* mFinalOutput{};
    int* mSourceToExpandedMap;
    int* mSelectedExpert;
    bool* mFinished{};
    int64_t mInterSize{};
    int64_t mTotalTokens{};
    int64_t mActiveRows{};

    bool mUseBias = true;
    bool mUseLora = false;

    bool mIsGated = false;
    int64_t mGatedMultiplier = 1;

    tensorrt_llm::ActivationType mActType = tensorrt_llm::ActivationType::Relu;
    MOEExpertScaleNormalizationMode mNormMode = MOEExpertScaleNormalizationMode::NONE;

    float mSparseMixerEpsilon = 0.2f;

    // Default this to true. This only matters for K>2, and so by doing this we will test the fused and unfused paths
    bool mUseDeterminsiticHopperReduce = true;

    // Disable this for long running tests to speed up runtime
    bool mIsLongTest = false;

    // If the test sets mOverrideSelectedConfig1 the BasicPermuteTest and *ParallelTests will use that instead of
    // looping over samples for the different architectures we support.
    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mOverrideSelectedConfig1 = std::nullopt;
    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mOverrideSelectedConfig2 = std::nullopt;

    // This is the actual tactic we use internally in runMoePermute
    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mInternalSelectedConfig1 = std::nullopt;
    std::optional<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> mInternalSelectedConfig2 = std::nullopt;

    // Keep to simple power of two so we can have tight bounds on precision for quantized modes
    float const mExpertWDiag1{0.5};
    float const mExpertWDiagGated{1};
    float const mExpertWDiag2{2};

    float mMaxInput{};

    template <class AllocType>
    AllocType* allocBuffer(size_t size)
    {
        size_t size_bytes = cute::ceil_div(size * sizeof_bits<AllocType>::value, 8);
        managed_buffers.emplace_back(mBufferManager->gpu(size_bytes));
        EXPECT_EQ(hipGetLastError(), hipSuccess) << "Error allocating buffer of size: " << size;
        AllocType* ptr = static_cast<AllocType*>(managed_buffers.back()->data());
        // Memset to an obviously incorrect value, so we detect any issues with uninitialised fields
        check_cuda_error(hipMemsetAsync(ptr, 0xD5, size_bytes, mStream->get()));
        return ptr;
    }

    bool checkSufficientTestMemory(int64_t num_tokens, int64_t hidden_size, int64_t num_experts, int64_t k)
    {
        this->managed_buffers.clear();             // Make sure all the previous buffers are freed
        check_cuda_error(hipDeviceSynchronize()); // Sync to make sure all previous operations are resolved

        // Calculate the size contributions for all the large buffers to check if the GPU has enough space
        bool const is_gated = tensorrt_llm::isGatedActivation(mActType);
        size_t const num_gemms = 2 + is_gated;
        bool const useDeepseek = false;

        // Expert weights
        size_t const weight_size
            = hidden_size * (hidden_size * 4) * num_experts * sizeof(WeightStorage) * num_gemms / WEIGHT_ELEM_PER_BYTE;
        // Workspace size
        size_t const workspace_size = this->mMoERunner.getWorkspaceSize(num_tokens, hidden_size, hidden_size * 4,
            num_experts, k, this->mActType, mNormMode, {}, mUseLora, useDeepseek);
        // The input/output buffers
        size_t const in_out_size = 2 * num_tokens * hidden_size * sizeof(DataType);

        // This should be correct to within 100MiB (on tests with 30GiB total)
        size_t const total_size = workspace_size + weight_size + in_out_size;

        size_t const memory_pool_free_mem_size = mBufferManager->memoryPoolFree();
        auto const [freeMem, totalMem] = tensorrt_llm::common::getDeviceMemoryInfo(false);
        float const freeMemBuffer = 0.9f; // Add some buffer so we aren't completely pushing the limits
        std::cout << "Free memory is: " << freeMem << ", memory pool size is: " << memory_pool_free_mem_size
                  << ", required memory is: " << total_size << ", device total memory capacity: " << totalMem
                  << std::endl;
        return (freeMem + memory_pool_free_mem_size) * freeMemBuffer >= total_size;
    }

    void initBuffersPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int64_t hidden_size, int64_t num_experts, int64_t k,
        std::vector<uint8_t> finished, MOEParallelismConfig parallelism_config)
    {
        managed_buffers.clear();

        mMoERunner.use_deterministic_hopper_reduce_ = k > 2 && mUseDeterminsiticHopperReduce;

        mHiddenSize = hidden_size;
        mInterSize = hidden_size * 4;
        mNumExperts = num_experts;
        mK = k;
        mIsGated = tensorrt_llm::isGatedActivation(mActType);
        mGatedMultiplier = mIsGated ? 2 : 1;
        auto const gated_inter = mInterSize * mGatedMultiplier;

        mTotalTokens = 0;

        std::vector<int64_t> h_seq_lens;
        h_seq_lens.push_back(0);
        for (auto& sequence : h_hidden_states)
        {
            assert(sequence.size() % hidden_size == 0);
            int64_t num_tokens = sequence.size() / hidden_size;
            h_seq_lens.emplace_back(h_seq_lens.back() + num_tokens);
            mTotalTokens += num_tokens;
        }

        bool const useDeepseek = false;
        size_t workspace_size = mMoERunner.getWorkspaceSize(mTotalTokens, mHiddenSize, mInterSize, mNumExperts, mK,
            mActType, mNormMode, parallelism_config, mUseLora, useDeepseek);

        auto const stream = mStream->get();

        mWorkspace = allocBuffer<char>(workspace_size);

        size_t const expert_matrix_size = mNumExperts * mHiddenSize * mInterSize;

        mRawExpertWeight1 = allocBuffer<DataType>(expert_matrix_size * mGatedMultiplier);
        mRawExpertWeight2 = allocBuffer<DataType>(expert_matrix_size);

        size_t const experts_per_node = mNumExperts / parallelism_config.ep_size;
        int const moe_parallel_size = parallelism_config.tp_size * parallelism_config.ep_size;

        mTpExpertScratchSize = expert_matrix_size * mGatedMultiplier / moe_parallel_size;
        mTpExpertScratchSize += expert_matrix_size / moe_parallel_size;

        mExpertBias1 = nullptr;
        mExpertBias2 = nullptr;
        if (mUseBias)
        {
            // Allow space for the slice of bias1 in the scratch
            mTpExpertScratchSize += experts_per_node * gated_inter / parallelism_config.tp_size;
            mExpertBias1 = allocBuffer<DataType>(mNumExperts * gated_inter);
            mExpertBias2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);

            check_cuda_error(hipMemsetAsync(mExpertBias1, 0x0, mNumExperts * gated_inter * sizeof(DataType), stream));
            check_cuda_error(hipMemsetAsync(mExpertBias2, 0x0, mNumExperts * mHiddenSize * sizeof(DataType), stream));
        }

        if constexpr (INT_QUANT)
        {
            mExpertWeight1 = allocBuffer<WeightStorage>(expert_matrix_size * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE);
            mExpertWeight2 = allocBuffer<WeightStorage>(expert_matrix_size / WEIGHT_ELEM_PER_BYTE);

            mTpExpertScratchSize += experts_per_node * gated_inter / parallelism_config.tp_size;
            mExpertIntScale1 = allocBuffer<DataType>(mNumExperts * gated_inter);
            mExpertIntScale2 = allocBuffer<DataType>(mNumExperts * mHiddenSize);
        }
        else if constexpr (FP4)
        {
            mExpertWeight1 = allocBuffer<WeightStorage>(expert_matrix_size * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE);
            mExpertWeight2 = allocBuffer<WeightStorage>(expert_matrix_size / WEIGHT_ELEM_PER_BYTE);

            size_t const padded_fc1_size = mNumExperts * mHiddenSize
                * cute::ceil_div(
                    mInterSize * mGatedMultiplier, TmaWarpSpecializedGroupedGemmInput::MinNumRowsAlignmentFP4)
                * TmaWarpSpecializedGroupedGemmInput::MinNumRowsAlignmentFP4;
            size_t const padded_fc2_size = mNumExperts * mInterSize
                * cute::ceil_div(mHiddenSize, TmaWarpSpecializedGroupedGemmInput::MinNumRowsAlignmentFP4)
                * TmaWarpSpecializedGroupedGemmInput::MinNumRowsAlignmentFP4;
            mFP4ScalingFactorsW1 = allocBuffer<ElementSF>(padded_fc1_size / FP4VecSize);
            mFP4ScalingFactorsW2 = allocBuffer<ElementSF>(padded_fc2_size / FP4VecSize);
        }
        else
        {
            mExpertWeight1 = mRawExpertWeight1;
            mExpertWeight2 = mRawExpertWeight2;
        }

        if constexpr (FP8 || FP4)
        {
            // FP4 uses the same logic as FP8 to generate the global scales
            mExpertFPXScale1 = allocBuffer<float>(mNumExperts);
            mExpertFPXScale2 = allocBuffer<float>(1);
            mExpertFPXScale3 = allocBuffer<float>(mNumExperts);

            if (FP4)
            {
                mExpertFP4ActGlobalScale1 = allocBuffer<float>(1);
                mExpertFP4WeightGlobalScale1 = allocBuffer<float>(mNumExperts);
                mExpertFP4WeightGlobalScale2 = allocBuffer<float>(mNumExperts);
            }

            EXPECT_NE(mMaxInput, 0.0f);
            initFPQuantScales(mMaxInput);
        }

        if (parallelism_config.tp_size > 1 || parallelism_config.ep_size > 1)
        {
            mTpExpertScratch = allocBuffer<DataType>(mTpExpertScratchSize);
        }

        mActiveRows = mTotalTokens;
        mFinished = nullptr;
        if (!finished.empty())
        {
            mFinished = allocBuffer<bool>(mTotalTokens);
            check_cuda_error(hipMemcpyAsync(
                mFinished, finished.data(), mTotalTokens * sizeof(bool), hipMemcpyHostToDevice, stream));
            static_assert(sizeof(bool) == sizeof(uint8_t), "Test assumes bool is interchangeable with uint8_t");
            mActiveRows = std::count(finished.begin(), finished.end(), 0);
        }

        mInputProbabilities = allocBuffer<float>(mTotalTokens * mNumExperts);
        mScaleProbs = allocBuffer<float>(mTotalTokens * mK);
        mInputTensor = allocBuffer<DataType>(mTotalTokens * mHiddenSize);
        mFinalOutput = allocBuffer<OutputType>(mTotalTokens * mHiddenSize);

        mSourceToExpandedMap = allocBuffer<int>(mTotalTokens * mK);
        mSelectedExpert = allocBuffer<int>(mTotalTokens * mK);

        auto* input_probs_ptr = mInputProbabilities;
        for (auto& sequence : h_router_results)
        {
            check_cuda_error(hipMemcpyAsync(
                input_probs_ptr, sequence.data(), sequence.size() * sizeof(float), hipMemcpyHostToDevice, stream));
            input_probs_ptr += sequence.size();
        }

        auto* hidden_states_ptr = mInputTensor;
        for (auto& sequence : h_hidden_states)
        {
            check_cuda_error(hipMemcpyAsync(hidden_states_ptr, sequence.data(), sequence.size() * sizeof(DataType),
                hipMemcpyHostToDevice, stream));
            hidden_states_ptr += sequence.size();
        }

        check_cuda_error(hipStreamSynchronize(stream));

        // Init the diagonals of our matrix, this will set to the scalar value
        initWeightsGated(
            mRawExpertWeight1, mHiddenSize, gated_inter, mExpertWDiag1, mExpertWDiagGated, mFP8WeightScalar1);
        initWeights(mRawExpertWeight2, mInterSize, mHiddenSize, mExpertWDiag2, mFP8WeightScalar2);

        if (mUseBias)
        {
            initBiasGated(mExpertBias1, gated_inter);
            initBias(mExpertBias2, mHiddenSize);
        }

        if constexpr (INT_QUANT)
        {
            cutlass_kernels::QuantType quant_type
                = INT4 ? cutlass_kernels::QuantType::W4_A16 : cutlass_kernels::QuantType::W8_A16;

            std::vector<size_t> shape1{(size_t) mNumExperts, (size_t) mHiddenSize, (size_t) gated_inter};
            std::vector<size_t> shape2{(size_t) mNumExperts, (size_t) mInterSize, (size_t) mHiddenSize};

            doIntQuant(quant_type, shape1, mRawExpertWeight1, mExpertIntScale1, mExpertWeight1);
            doIntQuant(quant_type, shape2, mRawExpertWeight2, mExpertIntScale2, mExpertWeight2);
        }

        check_cuda_error(hipStreamSynchronize(stream));
    }

    void doIntQuant(cutlass_kernels::QuantType quant_type, std::vector<size_t> shape, DataType* inputs,
        DataType* scales, uint8_t* outputs)
    {
        // Runs on the CPU, must be after stream sync
        if constexpr (INT_QUANT)
        {
            check_cuda_error(hipStreamSynchronize(mStream->get()));

            size_t elems = std::reduce(shape.begin(), shape.end(), 1, std::multiplies{});
            std::vector<int8_t> h_out(elems);
            std::vector<DataType> h_input(elems);
            std::vector<DataType> h_scales(shape[0] * shape[2]);

            check_cuda_error(hipMemcpy(h_input.data(), inputs, elems * sizeof(DataType), hipMemcpyDeviceToHost));

            cutlass_kernels::symmetric_quantize(h_out.data(), h_scales.data(), h_input.data(), shape, quant_type, true);

            check_cuda_error(hipMemcpy(
                outputs, h_out.data(), elems * sizeof(int8_t) / WEIGHT_ELEM_PER_BYTE, hipMemcpyHostToDevice));
            check_cuda_error(
                hipMemcpy(scales, h_scales.data(), h_scales.size() * sizeof(DataType), hipMemcpyHostToDevice));
        }
    }

    void doFP4Quant(DataType const* raw_weights, WeightStorage* quant_weights, float const* global_scales,
        ElementSF* scaling_factors, int in_shape, int out_shape, int num_experts)
    {
        int const mMultiProcessorCount = tensorrt_llm::common::getMultiProcessorCount();
        int padded_stride = cute::ceil_div(out_shape, TmaWarpSpecializedGroupedGemmInput::MinNumRowsAlignmentFP4)
            * TmaWarpSpecializedGroupedGemmInput::MinNumRowsAlignmentFP4;
        check_cuda_error(hipMemsetAsync(scaling_factors, 0x0,
            num_experts * padded_stride * cutlass::ceil_div(in_shape, FP4VecSize) * sizeof(ElementSF), mStream->get()));
        for (int i = 0; i < num_experts; i++)
        {
            auto* weight_start = raw_weights + i * in_shape * out_shape;
            auto* quant_weight_start = quant_weights + i * in_shape * out_shape / WEIGHT_ELEM_PER_BYTE;
            auto* scaling_factor_start
                = scaling_factors + i * (int64_t) padded_stride * cutlass::ceil_div(in_shape, FP4VecSize);

            invokeFP4Quantization(out_shape, in_shape, weight_start, global_scales + i,
                reinterpret_cast<int64_t*>(quant_weight_start), reinterpret_cast<int32_t*>(scaling_factor_start), false,
                mMultiProcessorCount, mStream->get());
        }
    }

    constexpr static float getFP8Scalar(float in)
    {
        if (FP8)
            return FP8_MAX / in;
        if (FP4)
            // We need to represent the block SF using FP8, so the largest value should be at most FP4_MAX * FP8_MAX
            // return FP8_MAX * FP4_MAX / in;
            // We carefully control precision in FP4. We want to avoid introducing any non-powers of two
            return 2.0f;
        return 1.0f;
    }

    void initFPQuantScales(float max_input)
    {
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        // Add shift to the max because we add an adjustment for each expert so they get different results.
        float maxW1 = applyExpertShift<IS_SCALED_TYPE>(
            mIsGated ? std::max(mExpertWDiag1, mExpertWDiagGated) : mExpertWDiag1, mNumExperts - 1, mNumExperts);
        float maxW2 = applyExpertShift<IS_SCALED_TYPE>(mExpertWDiag2, mNumExperts - 1, mNumExperts);
        // Weight scales are well-behaved powers of two so we use a power of two to improve our FP8 precision
        float scaleW1 = 256.f / maxW1;
        float scaleW2 = 256.f / maxW2;
        float scaleAct1 = getFP8Scalar(max_input);

        float maxFC1Output = calcMLPVal(max_input, mNumExperts - 1) / maxW2;
        float scaleAct2 = getFP8Scalar(maxFC1Output);

        ASSERT_NE(mExpertFPXScale1, nullptr);
        ASSERT_NE(mExpertFPXScale2, nullptr);
        ASSERT_NE(mExpertFPXScale3, nullptr);

        std::vector<float> scales_1;
        std::vector<float> scales_2;
        std::vector<float> scales_3;
        if (FP4)
        {
            std::vector<float> scale_global_w1(mNumExperts);
            std::vector<float> scale_global_w2(mNumExperts);

            std::vector<float> scales_0(1, scaleAct1);
            scales_1 = std::vector<float>(mNumExperts);
            scales_2 = std::vector<float>(1, scaleAct2);
            scales_3 = std::vector<float>(mNumExperts);

            for (int i = 0; i < mNumExperts; i++)
            {
                float maxW1 = applyExpertShift<IS_SCALED_TYPE>(
                    (mIsGated ? std::max(mExpertWDiag1, mExpertWDiagGated) : mExpertWDiag1), i, mNumExperts);
                float maxW2 = applyExpertShift<IS_SCALED_TYPE>(mExpertWDiag2, i, mNumExperts);
                float scaleW1 = getFP8Scalar(maxW1);
                float scaleW2 = getFP8Scalar(maxW2);
                scale_global_w1[i] = scaleW1;
                scale_global_w2[i] = scaleW2;
                // TODO Per expert scaling factors
                scales_1[i] = 1.f / (scaleAct1 * scaleW1);
                scales_3[i] = 1.f / (scaleAct2 * scaleW2);
            }

            ASSERT_NE(mExpertFP4ActGlobalScale1, nullptr);
            ASSERT_NE(mExpertFP4WeightGlobalScale1, nullptr);
            ASSERT_NE(mExpertFP4WeightGlobalScale2, nullptr);
            check_cuda_error(hipMemcpyAsync(mExpertFP4ActGlobalScale1, scales_0.data(),
                scales_0.size() * sizeof(float), hipMemcpyHostToDevice, mStream->get()));
            check_cuda_error(hipMemcpyAsync(mExpertFP4WeightGlobalScale1, scale_global_w1.data(),
                scale_global_w1.size() * sizeof(float), hipMemcpyHostToDevice, mStream->get()));
            check_cuda_error(hipMemcpyAsync(mExpertFP4WeightGlobalScale2, scale_global_w2.data(),
                scale_global_w2.size() * sizeof(float), hipMemcpyHostToDevice, mStream->get()));
        }
        else
        {
            mFP8WeightScalar1 = scaleW1;
            mFP8WeightScalar2 = scaleW2;
            scales_1 = std::vector<float>(mNumExperts, 1.f / (scaleW1 * scaleAct1));
            scales_2 = std::vector<float>(1, scaleAct2);
            scales_3 = std::vector<float>(mNumExperts, 1.f / (scaleW2 * scaleAct2));
        }

        check_cuda_error(hipMemcpyAsync(mExpertFPXScale1, scales_1.data(), scales_1.size() * sizeof(float),
            hipMemcpyHostToDevice, mStream->get()));
        check_cuda_error(hipMemcpyAsync(mExpertFPXScale2, scales_2.data(), scales_2.size() * sizeof(float),
            hipMemcpyHostToDevice, mStream->get()));
        check_cuda_error(hipMemcpyAsync(mExpertFPXScale3, scales_3.data(), scales_3.size() * sizeof(float),
            hipMemcpyHostToDevice, mStream->get()));

        check_cuda_error(hipStreamSynchronize(mStream->get()));
    }

    void resetOutBuffers()
    {
        auto stream = mStream->get();

        check_cuda_error(hipStreamSynchronize(stream));

        if (mTpExpertScratch)
            check_cuda_error(hipMemsetAsync(mTpExpertScratch, 0x0, mTpExpertScratchSize, stream));
        check_cuda_error(hipMemsetAsync(mFinalOutput, 0x0, mTotalTokens * mHiddenSize * sizeof(OutputType), stream));
        check_cuda_error(hipMemsetAsync(mSourceToExpandedMap, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mSelectedExpert, 0x0, sizeof(int) * mTotalTokens * mK, stream));
        check_cuda_error(hipMemsetAsync(mScaleProbs, 0x0, sizeof(float) * mTotalTokens * mK, stream));

        check_cuda_error(hipStreamSynchronize(stream));
    }

    void resizeRouterInputs(
        std::vector<std::vector<float>>& h_router_results, int64_t num_experts, int64_t num_tokens_per_seq)
    {
        for (int64_t i = 0; i < h_router_results.size(); i++)
        {
            auto& seq_routing = h_router_results[i];
            int64_t num_tokens = num_tokens_per_seq;
            auto hardcoded_experts = seq_routing.size() / num_tokens;
            ASSERT_EQ(seq_routing.size(), hardcoded_experts * num_tokens);
            if (num_experts > hardcoded_experts)
            {
                auto pos = seq_routing.begin() + hardcoded_experts;
                for (int64_t i = 0; i < num_tokens; i++, pos += num_experts)
                {
                    pos = seq_routing.insert(pos, num_experts - hardcoded_experts, 0);
                }
            }
            ASSERT_EQ(seq_routing.size(), num_experts * num_tokens);
        }
    }

    template <class T>
    auto populateTokens(std::vector<T>& hidden_states)
    {
        // Can't use FP8 param because we recurse with a different type
        if constexpr (std::is_same_v<T, SafeFP8>)
        {
            // Call the standard setup and then perform the quantization manually
            std::vector<OutputType> internal_states(hidden_states.size());
            populateTokens(internal_states);

            mMaxInput = *std::max_element(internal_states.begin(), internal_states.end());
            float scalar = getFP8Scalar(mMaxInput);
            std::transform(internal_states.begin(), internal_states.end(), hidden_states.begin(),
                [scalar](OutputType in) -> T { return static_cast<T>((float) in * scalar); });
            // Do the reverse transformation since we only have so much precision and this is a pretty broad range
            std::transform(hidden_states.begin(), hidden_states.end(), internal_states.begin(),
                [scalar](T in) -> OutputType { return static_cast<OutputType>(((float) in) / scalar); });
            return internal_states;
        }
        else if constexpr (FP4)
        {
            float const max_scale = 1.0f;
            mMaxInput = FP4_MAX * max_scale;
            // Excludes 0.75 as this causes increased quantization error
            std::array allowed_values{-6.f, -4.f, -3.f, -2.f, -1.5f, -1.f, 0.0f, 1.f, 1.5f, 2.0f, 3.0f, 4.0f, 6.0f};
            float scale = 1.f / 32.f;
            int stride = TmaWarpSpecializedGroupedGemmInput::BlockScaleVectorSize;
            for (int i = 0; i < hidden_states.size(); i += stride)
            {
                for (int j = 0; j < stride; j++)
                {
                    hidden_states[i + j] = allowed_values[(i / stride + j) % allowed_values.size()] * scale;
                }
                mMaxInput = std::max(mMaxInput, FP4_MAX * scale);
                scale *= 2.f;
                if (scale >= max_scale)
                {
                    scale = 1 / 32.f;
                }
            }
            return hidden_states;
        }
        else
        {
            // Generates numbers in increments of 1/max_order_of_magnitude in the range [0, 1)
            constexpr int max_order_of_magnitude = 256;
            std::vector<int> base(hidden_states.size());
            std::iota(base.begin(), base.end(), 0);
            // Lambda subtracts a small value so we have some < 0 to test the activation for negatives
            std::transform(base.begin(), base.end(), hidden_states.begin(),
                [l = hidden_states.size(), max_order_of_magnitude](auto a) {
                    return T(float(a % max_order_of_magnitude) / float(max_order_of_magnitude))
                        - T(4.f / max_order_of_magnitude);
                });
            mMaxInput = *std::max_element(hidden_states.begin(), hidden_states.end());
            return hidden_states;
        }
    }

    void runMoEPermute(std::vector<std::vector<DataType>> h_hidden_states,
        std::vector<std::vector<float>> h_router_results, int64_t hidden_size, int64_t num_experts, int64_t k,
        std::vector<uint8_t> finished = {}, MOEParallelismConfig parallelism_config = {})
    {
        initBuffersPermute(std::move(h_hidden_states), std::move(h_router_results), hidden_size, num_experts, k,
            finished, parallelism_config);
        runMoEPermute(parallelism_config);
    }

    auto getWeights(MOEParallelismConfig parallelism_config)
    {
        void* ep_scale_1 = (FP8 || FP4) ? (void*) mExpertFPXScale1 : (void*) mExpertIntScale1;
        void* ep_scale_2 = (FP8 || FP4) ? (void*) mExpertFPXScale2 : (void*) mExpertIntScale2;
        void* ep_scale_3 = (FP8 || FP4) ? mExpertFPXScale3 : nullptr;

        using SliceWeightType = std::conditional_t<FP4, DataType, WeightStorage>;
        // FP4 accesses the unquantized weight
        constexpr int SLICED_WEIGHT_ELEM_PER_BYTE = FP4 ? 1 : WEIGHT_ELEM_PER_BYTE;
        SliceWeightType* slice_weight_1{};
        SliceWeightType* slice_weight_2{};
        if constexpr (FP4)
        {
            slice_weight_1 = mRawExpertWeight1;
            slice_weight_2 = mRawExpertWeight2;
        }
        else
        {
            slice_weight_1 = mExpertWeight1;
            slice_weight_2 = mExpertWeight2;
        }

        // Handle the case with no parallelism to not require the extra alloc
        if (parallelism_config.tp_size == 1 && parallelism_config.ep_size == 1)
        {
            return std::tuple{(void*) slice_weight_1, (void*) slice_weight_2, mExpertBias1, mExpertBias2, ep_scale_1,
                ep_scale_2, ep_scale_3};
        }

        // Slice weights for EP
        size_t const gated_inter = mInterSize * mGatedMultiplier;
        size_t const experts_per_node = mNumExperts / parallelism_config.ep_size;
        size_t const weight_matrix_size = mHiddenSize * mInterSize * experts_per_node / SLICED_WEIGHT_ELEM_PER_BYTE;
        size_t const bias_fc1_size = gated_inter * experts_per_node;
        size_t const bias_fc2_size = mHiddenSize * experts_per_node;
        size_t const scale1_size = gated_inter * experts_per_node;
        size_t const scale2_size = mHiddenSize * experts_per_node;
        auto* weight1_ptr = slice_weight_1 + weight_matrix_size * mGatedMultiplier * parallelism_config.ep_rank;
        auto* weight2_ptr = slice_weight_2 + weight_matrix_size * parallelism_config.ep_rank;
        auto* bias1_ptr = mUseBias ? mExpertBias1 + bias_fc1_size * parallelism_config.ep_rank : nullptr;
        auto* bias2_ptr = mUseBias ? mExpertBias2 + bias_fc2_size * parallelism_config.ep_rank : nullptr;

        if (INT_QUANT)
        {
            ep_scale_1 = mExpertIntScale1 + scale1_size * parallelism_config.ep_rank;
            ep_scale_2 = mExpertIntScale2 + scale2_size * parallelism_config.ep_rank;
        }
        if constexpr (FP8 || FP4)
        {
            ep_scale_1 = mExpertFPXScale1 + experts_per_node * parallelism_config.ep_rank;
            ep_scale_3 = mExpertFPXScale3 + experts_per_node * parallelism_config.ep_rank;
        }

        // Slice weights for TP
        void* scale_1 = ep_scale_1;
        void* scale_2 = ep_scale_2;
        void* scale_3 = ep_scale_3;

        int const tp_size = parallelism_config.tp_size;
        int const tp_rank = parallelism_config.tp_rank;

        size_t const matrix_size = mHiddenSize * mInterSize / tp_size;
        size_t const gated_matrix_size = mHiddenSize * mInterSize * mGatedMultiplier / tp_size;
        size_t const row_size_inter = mInterSize / tp_size;

        auto* weight_1 = reinterpret_cast<SliceWeightType*>(mTpExpertScratch);
        auto* weight_2 = weight_1 + experts_per_node * gated_matrix_size / SLICED_WEIGHT_ELEM_PER_BYTE;
        auto* bias_1
            = reinterpret_cast<DataType*>(weight_2 + experts_per_node * matrix_size / SLICED_WEIGHT_ELEM_PER_BYTE);

        // 2D memcpy just the slices we care about
        // TODO Re-quantize here with matrices divided
        size_t const row_size_1 = matrix_size * sizeof(SliceWeightType) / SLICED_WEIGHT_ELEM_PER_BYTE;
        check_cuda_error(
            hipMemcpy2DAsync(weight_1, row_size_1, (uint8_t*) weight1_ptr + row_size_1 * tp_rank, row_size_1 * tp_size,
                row_size_1, experts_per_node * mGatedMultiplier, hipMemcpyDeviceToDevice, mStream->get()));

        size_t const row_size_2 = row_size_inter * sizeof(SliceWeightType) / SLICED_WEIGHT_ELEM_PER_BYTE;
        check_cuda_error(
            hipMemcpy2DAsync(weight_2, row_size_2, (uint8_t*) weight2_ptr + row_size_2 * tp_rank, row_size_2 * tp_size,
                row_size_2, experts_per_node * mHiddenSize, hipMemcpyDeviceToDevice, mStream->get()));

        if (mUseBias)
        {
            size_t const row_size_bias = row_size_inter * sizeof(DataType);
            check_cuda_error(hipMemcpy2DAsync(bias_1, row_size_bias, (uint8_t*) bias1_ptr + row_size_bias * tp_rank,
                row_size_bias * tp_size, row_size_bias, experts_per_node * mGatedMultiplier, hipMemcpyDeviceToDevice,
                mStream->get()));
        }

        if constexpr (INT_QUANT)
        {
            scale_2 = ep_scale_2;
            size_t const row_size_scale = row_size_inter * sizeof(DataType);
            check_cuda_error(hipMemcpy2DAsync(scale_1, row_size_scale,
                (uint8_t*) ep_scale_1 + row_size_scale * tp_rank, row_size_scale * tp_size, row_size_scale,
                experts_per_node * mGatedMultiplier, hipMemcpyDeviceToDevice, mStream->get()));
        }

        bias_1 = mUseBias ? bias_1 : nullptr;

        return std::tuple{(void*) weight_1, (void*) weight_2, bias_1, bias2_ptr, scale_1, scale_2, scale_3};
    }

    auto getFilteredConfigs(int sm)
    {
        auto tactics = mMoERunner.getTactics();
        if (sm == 89)
        {
            // Filter some unsupported configs for L40S
            auto it = std::remove_if(tactics.begin(), tactics.end(),
                [&](auto conf)
                {
                    using tensorrt_llm::cutlass_extensions::CutlassTileConfig;
                    auto checks = std::vector{
                        // Fail for BF16/FP16
                        conf.tile_config_sm80 == CutlassTileConfig::CtaShape128x128x64_WarpShape64x32x64,
                        conf.tile_config_sm80 == CutlassTileConfig::CtaShape64x128x64_WarpShape32x64x64
                            && conf.stages == 4,
                        // Fail for FP8
                        FP8 && conf.tile_config_sm80 == CutlassTileConfig::CtaShape16x256x128_WarpShape16x64x128
                            && conf.stages >= 3,
                    };

                    return std::any_of(checks.begin(), checks.end(), [](auto v) { return v; });
                });
            tactics.erase(it, tactics.end());
        }

        EXPECT_FALSE(tactics.empty());

        return tactics;
    }

    auto selectTacticsForArch(int sm)
    {
        bool is_tma_warp_specialized = sm >= 90 && !INT_QUANT;
        auto tactics = getFilteredConfigs(sm);
        auto it = std::find_if(tactics.begin(), tactics.end(),
            [is_tma_warp_specialized](auto& c) { return c.is_tma_warp_specialized == is_tma_warp_specialized; });
        if (it == tactics.end())
        {
            // Fall back to any tactic
            std::cout << "WARNING: Could not find config for sm version " << sm << std::endl;
            return std::pair{tactics[0], tactics[0]};
        }

        return std::pair(*it, *it);
    }

    using ConfigsToTestVec = std::vector<std::pair<tensorrt_llm::cutlass_extensions::CutlassGemmConfig,
        tensorrt_llm::cutlass_extensions::CutlassGemmConfig>>;

    auto getAllTileConfigsToTest()
    {
        if (mOverrideSelectedConfig1 && mOverrideSelectedConfig2)
        {
            return ConfigsToTestVec{std::pair{*mOverrideSelectedConfig1, *mOverrideSelectedConfig2}};
        }

        int sm = getSMVersion();
        ConfigsToTestVec tactics = {selectTacticsForArch(sm)};
        if (sm >= 90 && !FP8 && !FP4)
        {
            // SM90+ should also grab some configs for SM80 to test them
            tactics.push_back(selectTacticsForArch(80));
        }
        return tactics;
    }

    void runMoEPermute(MOEParallelismConfig parallelism_config)
    {
        // Clear the buffers to blank so we can assume zero if not written
        resetOutBuffers();

        auto [weight1_ptr, weight2_ptr, bias1_ptr, bias2_ptr, scale1_ptr, scale2_ptr, scale3_ptr]
            = getWeights(parallelism_config);

        auto stream = mStream->get();
        auto tactic1 = mInternalSelectedConfig1;
        auto tactic2 = mInternalSelectedConfig2;
        if (!tactic1)
        {
            int sm = getSMVersion();
            std::tie(tactic1, tactic2) = selectTacticsForArch(sm);
        }
        ASSERT_TRUE(tactic1.has_value());
        ASSERT_TRUE(tactic2.has_value());

        QuantParams quant_params;
        if constexpr (INT_QUANT)
        {
            ASSERT_TRUE(scale1_ptr && scale2_ptr);
            quant_params = QuantParams::Int(scale1_ptr, scale2_ptr);
        }
        else if (FP8)
        {
            ASSERT_TRUE(scale1_ptr && scale2_ptr && scale3_ptr);
            quant_params = QuantParams::FP8(static_cast<float const*>(scale1_ptr),
                static_cast<float const*>(scale2_ptr), static_cast<float const*>(scale3_ptr));
        }
        else if (FP4)
        {
            ASSERT_TRUE(mExpertFP4ActGlobalScale1);
            ASSERT_TRUE(mFP4ScalingFactorsW1 && mFP4ScalingFactorsW2);
            ASSERT_TRUE(scale1_ptr && scale2_ptr && scale3_ptr);
            quant_params = QuantParams::FP4(mExpertFP4ActGlobalScale1, mFP4ScalingFactorsW1,
                static_cast<float const*>(scale1_ptr), static_cast<float const*>(scale2_ptr), mFP4ScalingFactorsW2,
                static_cast<float const*>(scale3_ptr));
        }

        if constexpr (FP4)
        {
            // Dynamically quantize using the proper tp slice
            doFP4Quant(static_cast<DataType const*>(weight1_ptr), mExpertWeight1, mExpertFP4WeightGlobalScale1,
                mFP4ScalingFactorsW1, mHiddenSize, mGatedMultiplier * mInterSize / parallelism_config.tp_size,
                mNumExperts / parallelism_config.ep_size);
            doFP4Quant(static_cast<DataType const*>(weight2_ptr), mExpertWeight2, mExpertFP4WeightGlobalScale2,
                mFP4ScalingFactorsW2, mInterSize / parallelism_config.tp_size, mHiddenSize,
                mNumExperts / parallelism_config.ep_size);
            weight1_ptr = mExpertWeight1;
            weight2_ptr = mExpertWeight2;
        }

        LoraParams lora_params;
        BlockScaleParams deepseekParams{};
        bool const useDeepseek = false;

        mMoERunner.setTactic(tactic1, tactic2);
        mMoERunner.runMoe(mInputTensor, mInputProbabilities, weight1_ptr, bias1_ptr, mActType, weight2_ptr, bias2_ptr,
            quant_params, mTotalTokens, mHiddenSize, mInterSize / parallelism_config.tp_size, mNumExperts, mK,
            mWorkspace, mFinalOutput, mFinished, mActiveRows, mScaleProbs, mSourceToExpandedMap, mSelectedExpert,
            mSparseMixerEpsilon, parallelism_config, mNormMode, mUseLora, lora_params, useDeepseek, deepseekParams,
            stream);

        check_cuda_error(hipStreamSynchronize(stream));
    }

    template <class T>
    std::vector<T> getDataFromDevice(T const* in, size_t length)
    {
        std::vector<T> data(length);

        auto const stream = mStream->get();
        check_cuda_error(hipMemcpyAsync(data.data(), in, length * sizeof(T), hipMemcpyDeviceToHost, stream));
        check_cuda_error(hipStreamSynchronize(mStream->get()));

        return data;
    }

    auto maskSelectedExpertsForTP(std::vector<int> const& vector, int tp_size, int tp_rank)
    {
        std::vector<int> result;
        int num_experts_per_node = mNumExperts / tp_size;
        std::transform(vector.begin(), vector.end(), std::back_inserter(result),
            [=](int entry)
            {
                if (entry >= num_experts_per_node * tp_rank && entry < num_experts_per_node * (tp_rank + 1))
                    return entry;
                return (int) mNumExperts + entry;
            });
        return result;
    }

    void debugPrint()
    {
#define PRINT_CAST(array, size, cast)                                                                                  \
    do                                                                                                                 \
        if (array)                                                                                                     \
        {                                                                                                              \
            auto data = getDataFromDevice(array, size);                                                                \
            std::cout << #array << ": ";                                                                               \
            for (auto v : data)                                                                                        \
            {                                                                                                          \
                if (cast(v))                                                                                           \
                    std::cout << cast(v) << ", ";                                                                      \
                else                                                                                                   \
                    std::cout << "., ";                                                                                \
            }                                                                                                          \
            std::cout << std::endl;                                                                                    \
        }                                                                                                              \
    while (0)
#define PRINT(array, size) PRINT_CAST(array, size, )

        using WeightPrintType = std::conditional_t<INT_QUANT, uint8_t, WeightStorage>;
        PRINT_CAST((WeightPrintType*) mExpertWeight1,
            mNumExperts * mHiddenSize * mInterSize * mGatedMultiplier / WEIGHT_ELEM_PER_BYTE, float);
        PRINT_CAST(
            (WeightPrintType*) mExpertWeight2, mNumExperts * mHiddenSize * mInterSize / WEIGHT_ELEM_PER_BYTE, float);
        // PRINT_CAST(mRawExpertWeight1, mNumExperts * mHiddenSize * mInterSize * mGatedMultiplier, float);
        // PRINT_CAST(mRawExpertWeight2, mNumExperts * mHiddenSize * mInterSize, float);
        PRINT_CAST(mExpertBias1, mNumExperts * mInterSize * mGatedMultiplier, float);
        PRINT_CAST(mExpertBias2, mNumExperts * mHiddenSize, float);
        PRINT_CAST(mExpertIntScale1, mNumExperts * mInterSize * mGatedMultiplier, float);
        PRINT_CAST(mExpertIntScale2, mNumExperts * mHiddenSize, float);
        PRINT(mFinalOutput, mTotalTokens * mHiddenSize);
        PRINT_CAST((uint8_t*) mFinished, mTotalTokens, (int) );
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT(mScaleProbs, mTotalTokens * mK);
        PRINT(mInputProbabilities, mTotalTokens * mNumExperts);
        PRINT_CAST(mInputTensor, mTotalTokens * mHiddenSize, float);
        PRINT(mSourceToExpandedMap, mTotalTokens * mK);
        PRINT(mSelectedExpert, mTotalTokens * mK);

#undef PRINT_CAST
#undef PRINT
    }

    template <class T>
    T actfn(T in)
    {
        if (mActType == tensorrt_llm::ActivationType::Identity)
            return in;
        if (mActType == tensorrt_llm::ActivationType::Relu)
            return std::max(in, T(0.0f));
        if (mActType == tensorrt_llm::ActivationType::Gelu || mActType == tensorrt_llm::ActivationType::Geglu)
            return (std::erf(float(in) * float(sqrt(0.5))) + 1) * 0.5f * float(in);
        if (mActType == tensorrt_llm::ActivationType::Silu || mActType == tensorrt_llm::ActivationType::Swiglu)
        {
            return (float(in) / (1.f + std::exp(-(in))));
        }
        assert(false);
        return in;
    }

    float calcMLPVal(float input, int expert_id, bool final_bias = false)
    {
        if (expert_id >= mNumExperts)
            return 0;

        float w1_bias = mUseBias ? expert_id : 0.f;
        float activated = 0;
        if (mIsGated)
        {
            float scalar = applyExpertShift<IS_SCALED_TYPE>(mExpertWDiag1, expert_id, mNumExperts);
            float fc1 = input * scalar + w1_bias;

            float gated_scalar = applyExpertShift<IS_SCALED_TYPE>(mExpertWDiagGated, expert_id, mNumExperts);
            float gated_bias = mUseBias ? w1_bias + 1.f : 0.f;
            float gate = input * gated_scalar + gated_bias;

            activated = fc1 * actfn(gate);
        }
        else
        {
            float scalar = applyExpertShift<IS_SCALED_TYPE>(mExpertWDiag1, expert_id, mNumExperts);
            float fc1 = input * scalar + w1_bias;
            activated = actfn(fc1);
        }

        EXPECT_TRUE(mUseBias || !final_bias);
        float result = activated * applyExpertShift<IS_SCALED_TYPE>(mExpertWDiag2, expert_id, mNumExperts)
            + (float) (final_bias ? expert_id : 0);
        return result;
    }

    float calcMLPValWithFinalBias(float input, int expert_id)
    {
        return calcMLPVal(input, expert_id, mUseBias);
    }

    // NOTE This is a useful function for debugging routing failures. But you need to know the exact offset of
    //   this info in the workspace so having a test depend on something so internal is suboptimal
    //
    // void comparePermuted(const std::vector<int>& expected_experts, const std::vector<int>& expected_permutation,
    //     const std::vector<DataType>& input_data)
    //{
    //     auto states = getDataFromDevice(magic incantation into workspace, mTotalTokens * mK * mHiddenSize);
    //
    //    // Loop for the number of times each token is duplicated
    //    for (int k_idx = 0; k_idx < mK; k_idx++)
    //    {
    //        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
    //        {
    //            // Permutation has the position of the first copy of all token,
    //            // followed by the position of the second copy of all tokens etc.
    //            const int64_t permuted_position = expected_permutation[k_idx * mTotalTokens + token_id];
    //
    //            // Expected experts has all the selected experts for token one,
    //            // followed by all the selected experts for token two etc.
    //            const int64_t expert_id = expected_experts[token_id * mK + k_idx];
    //
    //            // Compare the copied tokens with the projection applied
    //            for (int64_t hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
    //            {
    //                auto ref = calcMLPVal(input_data[token_id * mHiddenSize + hidden_id], expert_id);
    //                auto actual = states[permuted_position * mHiddenSize + hidden_id];
    //                ASSERT_NEAR(ref, actual, getTolerance(ref))
    //                    << "Incorrect value at position: mK: " << k_idx << ", token: " << token_id
    //                    << ", permuted dest: " << permuted_position << ", expert id: " << expert_id
    //                    << ", hidden id: " << hidden_id;
    //            }
    //        }
    //    }
    //}

    std::vector<float> softmax(std::vector<float> const& expected_probs)
    {
        std::vector<float> softmax;
        // All values we test are 0-1 so we can skip the normalization step
        std::transform(expected_probs.begin(), expected_probs.end(), std::back_inserter(softmax),
            [&](float const in) -> float
            {
                auto res = exp(in);
                return res;
            });

        for (int64_t token = 0; token < softmax.size(); token += mNumExperts)
        {
            auto start = softmax.begin() + token;
            auto end = start + mNumExperts;
            auto sum = std::accumulate(start, end, 0.f);
            std::transform(start, end, start, [=](auto in) { return in / sum; });
        }

        return softmax;
    }

    void renormScales(float* probs, int const* experts)
    {
        if (!moeRoutingNeedsRenorm(mNormMode))
            return;
        float sum = 0;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            sum += probs[experts[k_idx]];
        }
        float norm_factor = 1.0f / sum;
        for (int k_idx = 0; k_idx < mK; k_idx++)
        {
            probs[experts[k_idx]] *= norm_factor;
        }
    }

    float sparseMixer(std::vector<float> logits, int token_idx, int k_idx, int expected_expert)
    {
        EXPECT_LE(mK, 2);
        EXPECT_LT(k_idx, mK);
        EXPECT_LT(token_idx * mNumExperts, logits.size());
        EXPECT_LE((token_idx + 1) * mNumExperts, logits.size());

        auto start_it = logits.begin() + token_idx * mNumExperts;
        auto end_it = logits.begin() + (token_idx + 1) * mNumExperts;

        // Mask old maxes and get the kth largest
        auto max_it = end_it;
        for (int i = 0; i <= k_idx; i++)
        {
            max_it = std::max_element(start_it, end_it);
            if (i != k_idx)
            {
                EXPECT_NE(max_it, end_it);
                *max_it = -INFINITY;
            }
        }

        EXPECT_EQ((max_it - start_it), expected_expert)
            << "Expected token " << token_idx << " k_idx " << k_idx << " to select expert " << expected_expert;

        std::vector<float> masked;
        std::transform(start_it, end_it, std::back_inserter(masked),
            [this, max_it](auto val)
            {
                float mask_value = (*max_it - val) / max(abs(val), *max_it);
                return (mask_value > 2 * mSparseMixerEpsilon) ? -INFINITY : val;
            });
        auto output_probs = softmax(masked);
        return output_probs[expected_expert];
    }

    template <class T>
    [[nodiscard]] auto repeat(std::vector<T> const& vector, int64_t repetitions)
    {
        return repeat_blocks(vector, vector.size(), repetitions);
    }

    template <class T>
    [[nodiscard]] auto repeat_blocks(std::vector<T> const& vector, int64_t block_size, int64_t repetitions)
    {
        std::vector<T> output;
        output.reserve(vector.size() * repetitions);
        for (int64_t block = 0; block < vector.size(); block += block_size)
        {
            for (int rep = 0; rep < repetitions; rep++)
            {
                output.insert(output.end(), vector.begin() + block, vector.begin() + block + block_size);
            }
        }
        return output;
    }

    template <class T>
    [[nodiscard]] auto flatten(std::vector<std::vector<T>> const& vector)
    {
        std::vector<T> output;
        for (auto& v : vector)
        {
            output.insert(output.end(), v.begin(), v.end());
        }
        return output;
    }

    void compareSoftmax(std::vector<int> const& expected_experts,
        std::vector<std::vector<float>> const& expected_probs_unflatten, std::vector<float> scale_probs = {})
    {
        auto expected_probs = flatten(expected_probs_unflatten);
        ASSERT_EQ(expected_experts.size() / mK, expected_probs.size() / mNumExperts);
        if (scale_probs.empty())
            scale_probs = getDataFromDevice(mScaleProbs, mTotalTokens * mK);
        auto softmax_probs = moeRoutingNeedsSoftmax(mNormMode) ? softmax(expected_probs) : expected_probs;

        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
        {
            renormScales(&softmax_probs[token_id * mNumExperts], &expected_experts[token_id * mK]);

            for (int k_idx = 0; k_idx < mK; k_idx++)
            {
                int selected_expert = expected_experts[token_id * mK + k_idx];
                if (selected_expert < mNumExperts) // Ignore 'finished' values
                {
                    float expected_value = softmax_probs[token_id * mNumExperts + selected_expert];
                    if (mNormMode == tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::SPARSE_MIXER)
                    {
                        expected_value = sparseMixer(expected_probs, token_id, k_idx, selected_expert);
                    }

                    ASSERT_NEAR(expected_value, scale_probs[token_id * mK + k_idx], getTolerance())
                        << "Scales mismatched for token: " << token_id << " k: " << k_idx
                        << " selected_expert: " << selected_expert;
                }
            }
        }
    }

    void compareFinal(std::vector<int> const& expected_experts,
        std::vector<std::vector<float>> const& expected_probs_unflatten,
        std::vector<std::vector<OutputType>> const& input_data_unflatten, std::vector<OutputType> final_results = {})
    {
        auto expected_probs = flatten(expected_probs_unflatten);
        auto input_data = flatten(input_data_unflatten);
        ASSERT_EQ(expected_experts.size() / mK, expected_probs.size() / mNumExperts);
        ASSERT_EQ(expected_experts.size() / mK, input_data.size() / mHiddenSize);
        if (final_results.empty())
            final_results = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);

        auto softmax_probs = moeRoutingNeedsSoftmax(mNormMode) ? softmax(expected_probs) : expected_probs;
        for (int64_t token_id = 0; token_id < mTotalTokens; token_id++)
        {
            renormScales(&softmax_probs[token_id * mNumExperts], &expected_experts[token_id * mK]);

            for (int64_t hidden_id = 0; hidden_id < mHiddenSize; hidden_id++)
            {
                float sum = 0.0f;
                // Loop for the number of times each token is duplicated
                for (int k_idx = 0; k_idx < mK; k_idx++)
                {
                    int selected_expert = expected_experts[token_id * mK + k_idx];

                    float scale_value = softmax_probs[token_id * mNumExperts + selected_expert];
                    if (mNormMode == tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::SPARSE_MIXER)
                    {
                        scale_value = sparseMixer(expected_probs, token_id, k_idx, selected_expert);
                    }

                    float final_value = float(calcMLPValWithFinalBias(
                        static_cast<float>(input_data[token_id * mHiddenSize + hidden_id]), selected_expert));
                    sum += final_value * scale_value;
                }

                ASSERT_NEAR(OutputType{sum}, final_results[token_id * mHiddenSize + hidden_id], getTolerance(sum))
                    << "Incorrect final value at for token: " << token_id << " offset: " << hidden_id;
            }
        }
    }

    void BasicPermuteTest(
        int k = 1, int64_t hidden_size = DEFAULT_HIDDEN_SIZE, int64_t num_experts = 4, int64_t batch_size = 1);

    std::vector<int> calcPermuteMapExpertParallel(std::vector<int> const& expected_experts);

    void ExpertParallelTest(
        int k = 1, int64_t hidden_size = DEFAULT_HIDDEN_SIZE, int64_t num_experts = 4, int64_t batch_size = 1)
    {
        // 2 experts per rank
        ParallelismTest(k, 1, num_experts / 2, hidden_size, num_experts, batch_size);
        // 1 expert per rank
        ParallelismTest(k, 1, num_experts, hidden_size, num_experts, batch_size);
    }

    void TensorParallelTest(
        int k = 1, int64_t hidden_size = DEFAULT_HIDDEN_SIZE, int64_t num_experts = 4, int64_t batch_size = 1)
    {
        ParallelismTest(k, 2, 1, hidden_size, num_experts, batch_size);
        ParallelismTest(k, 4, 1, hidden_size, num_experts, batch_size);
        ParallelismTest(k, 8, 1, hidden_size, num_experts, batch_size);
    }

    void MixedParallelTest(
        int k = 1, int64_t hidden_size = DEFAULT_HIDDEN_SIZE, int64_t num_experts = 4, int64_t batch_size = 1)
    {
        // 2 experts per rank
        ParallelismTest(k, 2, num_experts / 2, hidden_size, num_experts, batch_size);
        ParallelismTest(k, 8, num_experts / 2, hidden_size, num_experts, batch_size);

        // 1 expert per rank
        ParallelismTest(k, 2, num_experts, hidden_size, num_experts, batch_size);
        ParallelismTest(k, 8, num_experts, hidden_size, num_experts, batch_size);
    }

    void ParallelismTest(int k = 1, int tp_size = 4, int ep_size = 2, int64_t hidden_size = DEFAULT_HIDDEN_SIZE,
        int64_t num_experts = 4, int64_t batch_size = 1);
};

template <class WeightParams>
using LargeMixtureOfExpertsTest = MixtureOfExpertsTest<WeightParams>;

template <class DataType_, class WeightType_ = DataType_, class OutputType_ = DataType_>
struct WeightParams
{
    using DataType = DataType_;
    using WeightType = WeightType_;
    using OutputType = OutputType_;
};

// TODO Fix int quantized
using Types = ::testing::Types<
#ifdef ENABLE_BF16
    WeightParams<__hip_bfloat16>,
#endif
#ifdef ENABLE_FP8
    WeightParams<SafeFP8, SafeFP8, half>,
#endif
#ifdef ENABLE_FP4
    WeightParams<SafeFP4, SafeFP4, half>,
#endif
    WeightParams<half>, WeightParams<float>

    //, WeightParams<half, uint8_t>, WeightParams<half, cutlass::uint4b_t>

    >;
TYPED_TEST_SUITE(MixtureOfExpertsTest, Types);

// Have a separate test with only FP8 and half data type because this test is long
using LargeTestTypes = ::testing::Types<
#ifdef ENABLE_FP8
    WeightParams<SafeFP8, SafeFP8, half>,
#endif
    WeightParams<half>>;
TYPED_TEST_SUITE(LargeMixtureOfExpertsTest, LargeTestTypes);

template <class TypeParam_>
BufferManager::CudaStreamPtr MixtureOfExpertsTest<TypeParam_>::mStream{};
template <class TypeParam_>
std::unique_ptr<BufferManager> MixtureOfExpertsTest<TypeParam_>::mBufferManager{};
template <class TypeParam_>
int MixtureOfExpertsTest<TypeParam_>::mDeviceCount{};

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::BasicPermuteTest(
    int k, int64_t hidden_size, int64_t num_experts, int64_t batch_size)
{
    if constexpr (FP8 || FP4)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    if (FP4)
    {
        if (mActType != tensorrt_llm::ActivationType::Relu)
        {
            // FP4 has far too little precision to get any sort of consistency with non-relu actfn
            GTEST_SKIP();
            return;
        }
    }

    auto test_archs = getAllTileConfigsToTest();
    for (auto [gemm1, gemm2] : test_archs)
    {
        mInternalSelectedConfig1 = gemm1;
        mInternalSelectedConfig2 = gemm2;

        //    int64_t num_experts = 4;
        int64_t numTokensInSeq = 3;

        // Input data for each sequence
        std::vector<DataType> hidden_states(hidden_size * numTokensInSeq);
        auto raw_unquant_states = populateTokens(hidden_states);

        std::vector<float> probs = {
            0.5, 0.1, 0.25, 0.15,   //
            0.03, 0.2, 0.07, 0.7,   //
            0.25, 0.21, 0.35, 0.19, //
        };

        std::vector<std::vector<DataType>> hidden_input(batch_size, hidden_states);
        std::vector<std::vector<float>> router_input(batch_size, probs);
        std::vector<std::vector<OutputType>> raw_unquant_input(batch_size, raw_unquant_states);
        resizeRouterInputs(router_input, num_experts, numTokensInSeq);

        runMoEPermute(hidden_input, router_input, hidden_size, num_experts, k);
        bool should_be_deterministic = mUseDeterminsiticHopperReduce || mK < 3 || getSMVersion() < 90;
        if (should_be_deterministic && !mIsLongTest)
        {
            auto first_iter = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);
            runMoEPermute(hidden_input, router_input, hidden_size, num_experts, k);
            auto second_iter = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);
            ASSERT_TRUE(std::equal(first_iter.begin(), first_iter.end(), second_iter.begin()))
                << "Running permute twice does not generate the same results";
        }

        std::vector<int> expected_experts{0, 3, 2};
        if (k == 2)
            expected_experts = {0, 2, 3, 1, 2, 0};
        else if (k == 3)
            expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};

        expected_experts = repeat(expected_experts, batch_size);

        auto selected_expert = getDataFromDevice(mSelectedExpert, mTotalTokens * k);
        EXPECT_EQ(selected_expert, expected_experts);

        auto proj_map = getDataFromDevice(mSourceToExpandedMap, mTotalTokens * k);
        // This is the final position of:
        // Token 1 Expert 1, T2E1, T3E1, T1E2, T2E2, T3E2
        std::vector<int> permute_map{0, 2, 1};
        if (k == 2)
            permute_map = {0, 5, 4, 3, 2, 1};
        if (k == 3)
            permute_map = {0, 8, 6, 4, 2, 1, 7, 5, 3};
        // For batch size > 1 we should just calculate the map from the experts
        if (batch_size > 1)
            permute_map = calcPermuteMapExpertParallel(expected_experts);
        ASSERT_EQ(permute_map, proj_map);
        compareSoftmax(selected_expert, router_input);
        compareFinal(selected_expert, router_input, raw_unquant_input);
    }
}

TYPED_TEST(MixtureOfExpertsTest, Permute)
{
    this->BasicPermuteTest();
}

TYPED_TEST(MixtureOfExpertsTest, PermuteK2)
{
    this->BasicPermuteTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteK3)
{
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSweepBatchSizes)
{
    this->mIsLongTest = true;
    for (int batch_size : {2, 8, 15, 19, 64, 73, 256})
    {
        this->BasicPermuteTest(1, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);
        this->BasicPermuteTest(2, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);
        this->BasicPermuteTest(3, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);
    }
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSweepBatchSizesGeglu)
{
    this->mIsLongTest = true;
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    for (int batch_size : {2, 8, 15, 19, 64, 73, 256})
    {
        this->BasicPermuteTest(1, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);
        this->BasicPermuteTest(2, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);
        this->BasicPermuteTest(3, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);
    }
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNoBias)
{
    this->mUseBias = false;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteRenormalization)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSparseMixer)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::SPARSE_MIXER;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteGelu)
{
    this->mActType = tensorrt_llm::ActivationType::Gelu;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSilu)
{
    this->mActType = tensorrt_llm::ActivationType::Silu;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteDeviceLimited)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::DEVICE_LIMITED;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteDeviceLimitedRenorm)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::DEVICE_LIMITED_RENORM;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteGeglu)
{
    this->mActType = tensorrt_llm::ActivationType::Geglu;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSwiglu)
{
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    this->BasicPermuteTest();
    this->BasicPermuteTest(2);
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNonDeterministic)
{
    this->mUseDeterminsiticHopperReduce = false;
    // Just test case 3, cases 1&2 always use the fused paths
    this->BasicPermuteTest(3);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteVerySmall)
{
    for (int i = 1; i <= 3; i++)
    {
        this->BasicPermuteTest(1, this->MINIMUM_ALIGNMENT * i);
        this->BasicPermuteTest(2, this->MINIMUM_ALIGNMENT * i);
        this->BasicPermuteTest(3, this->MINIMUM_ALIGNMENT * i);
    }
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNonPowerOfTwo)
{
    this->BasicPermuteTest(1, this->DEFAULT_HIDDEN_SIZE, 10);
    this->BasicPermuteTest(2, this->DEFAULT_HIDDEN_SIZE, 10);
    this->BasicPermuteTest(3, this->DEFAULT_HIDDEN_SIZE, 10);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNonPowerOfTwoRenorm)
{
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->BasicPermuteTest(1, this->DEFAULT_HIDDEN_SIZE, 10);
    this->BasicPermuteTest(2, this->DEFAULT_HIDDEN_SIZE, 10);
    this->BasicPermuteTest(3, this->DEFAULT_HIDDEN_SIZE, 10);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteNonPowerOfTwoSwiglu)
{
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    this->BasicPermuteTest(1, this->DEFAULT_HIDDEN_SIZE, 10);
    this->BasicPermuteTest(2, this->DEFAULT_HIDDEN_SIZE, 10);
    this->BasicPermuteTest(3, this->DEFAULT_HIDDEN_SIZE, 10);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteManyExperts)
{
    /* This test is very slow. Only do one k value */
    this->BasicPermuteTest(2, this->MINIMUM_ALIGNMENT, 512);
}

TYPED_TEST(MixtureOfExpertsTest, PermuteSwigluVerySmall)
{
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    for (int i = 1; i <= 3; i++)
    {
        this->BasicPermuteTest(1, this->MINIMUM_ALIGNMENT * i);
        this->BasicPermuteTest(2, this->MINIMUM_ALIGNMENT * i);
        this->BasicPermuteTest(3, this->MINIMUM_ALIGNMENT * i);
    }
}

TYPED_TEST(MixtureOfExpertsTest, PermuteMixtral8x7b)
{
    this->mIsLongTest = true;
    this->mUseBias = false;
    this->mActType = tensorrt_llm::ActivationType::Swiglu;
    this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;
    this->BasicPermuteTest(2, 4096, 8);
}

template <class TypeParam_>
std::vector<int> MixtureOfExpertsTest<TypeParam_>::calcPermuteMapExpertParallel(
    std::vector<int> const& expected_experts)
{
    std::vector<int> map(expected_experts.size());
    auto getInterleavedIndex = [this](int i) { return (i % mK) * mTotalTokens + i / mK; };
    int map_idx = 0;
    for (int expert = 0; expert < mNumExperts * 2; expert++)
    {
        for (int i = 0; i < map.size(); i++)
        {
            if (expected_experts[i] == expert)
                map[getInterleavedIndex(i)] = map_idx++;
        }
    }

    return map;
}

template <class TypeParam_>
void MixtureOfExpertsTest<TypeParam_>::ParallelismTest(
    int k, int tp_size, int ep_size, int64_t hidden_size, int64_t num_experts, int64_t batch_size)
{
    if (FP8 || FP4)
    {
        // TODO Remove this when bias + FP8 is supported
        mUseBias = false;
    }

    if (FP4)
    {
        if (mActType != tensorrt_llm::ActivationType::Relu)
        {
            // FP4 has far too little precision to get any sort of consistency with non-relu actfn
            GTEST_SKIP();
            return;
        }
    }

    ASSERT_LE(ep_size, num_experts);
    if (tp_size == 1)
    {
        // Only the first 4 experts are ever used. They should be split across at least 2 ranks
        ASSERT_LT(num_experts / ep_size, 4)
            << "Expert parallelism must have less than 4 experts per rank or the test is ineffective";
    }

    auto test_archs = getAllTileConfigsToTest();
    for (auto [gemm1, gemm2] : test_archs)
    {
        mInternalSelectedConfig1 = gemm1;
        mInternalSelectedConfig2 = gemm2;

        int64_t numTokensInSeq = 3;

        std::vector<DataType> hidden_states(hidden_size * numTokensInSeq);
        auto raw_unquant_states = populateTokens(hidden_states);

        std::vector<float> probs = {
            0.5, 0.1, 0.25, 0.15,   //
            0.03, 0.2, 0.07, 0.7,   //
            0.25, 0.21, 0.35, 0.19, //
        };

        std::vector<std::vector<DataType>> hidden_input(batch_size, hidden_states);
        std::vector<std::vector<float>> router_input(batch_size, probs);
        std::vector raw_unquant_input(batch_size, raw_unquant_states);
        resizeRouterInputs(router_input, num_experts, numTokensInSeq);

        std::vector<int> expected_experts{0, 3, 2};
        if (k == 2)
            expected_experts = {0, 2, 3, 1, 2, 0};
        else if (k == 3)
            expected_experts = {0, 2, 3, 3, 1, 2, 2, 0, 1};
        expected_experts = repeat(expected_experts, batch_size);
        std::vector<OutputType> results(hidden_states.size() * batch_size, 0);
        for (int i = 0; i < tp_size; i++)
        {
            for (int j = 0; j < ep_size; j++)
            {
                if (i == 0 && j == 0)
                {
                    // Only need to init the inputs on the first iteration
                    runMoEPermute(hidden_input, router_input, hidden_size, num_experts, k, {},
                        MOEParallelismConfig{tp_size, i, ep_size, j});
                    bool should_be_deterministic = mUseDeterminsiticHopperReduce || mK < 3 || getSMVersion() < 90;
                    if (should_be_deterministic && !mIsLongTest)
                    {
                        auto first_iter = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);
                        runMoEPermute(hidden_input, router_input, hidden_size, num_experts, k, {},
                            MOEParallelismConfig{tp_size, i, ep_size, j});
                        auto second_iter = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);
                        ASSERT_TRUE(std::equal(first_iter.begin(), first_iter.end(), second_iter.begin()))
                            << "Running permute a second time does not generate the same results";
                    }
                }
                else
                {
                    runMoEPermute(MOEParallelismConfig{tp_size, i, ep_size, j});
                    bool should_be_deterministic = mUseDeterminsiticHopperReduce || mK < 3 || getSMVersion() < 90;
                    if (should_be_deterministic && !mIsLongTest)
                    {
                        auto first_iter = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);
                        runMoEPermute(MOEParallelismConfig{tp_size, i, ep_size, j});
                        auto second_iter = getDataFromDevice(mFinalOutput, mTotalTokens * mHiddenSize);
                        ASSERT_TRUE(std::equal(first_iter.begin(), first_iter.end(), second_iter.begin()))
                            << "Running permute a second time does not generate the same results";
                    }
                }

                auto selected_expert = getDataFromDevice(mSelectedExpert, mTotalTokens * k);
                // Experts should only be selected when we are on the right node
                // Note the index is [0,num_experts_per_node), so we offset the experts by the start for this node
                int const start_expert = j * (mNumExperts / ep_size);
                std::transform(selected_expert.begin(), selected_expert.end(), selected_expert.begin(),
                    [&](int val) { return val >= mNumExperts ? val : val + start_expert; });
                auto masked_expected_experts = maskSelectedExpertsForTP(expected_experts, ep_size, j);
                ASSERT_EQ(selected_expert, masked_expected_experts);

                auto proj_map = getDataFromDevice(mSourceToExpandedMap, mTotalTokens * k);
                auto permute_map = calcPermuteMapExpertParallel(masked_expected_experts);
                ASSERT_EQ(permute_map, proj_map) << "Iteration " << i << " " << j << " batch size " << batch_size;
                compareSoftmax(expected_experts, router_input);

                // Do the final reduce
                auto iter_results = getDataFromDevice(mFinalOutput, mTotalTokens * hidden_size);
                std::transform(
                    iter_results.cbegin(), iter_results.cend(), results.cbegin(), results.begin(), std::plus<>{});
            }
        }

        compareFinal(expected_experts, router_input, raw_unquant_input, results);
    }
}

#define PARALLEL_TEST_SUITE(ParallelismType)                                                                           \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType)                                                                  \
    {                                                                                                                  \
        this->ParallelismType##Test();                                                                                 \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##K2)                                                              \
    {                                                                                                                  \
        this->ParallelismType##Test(2);                                                                                \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##K3)                                                              \
    {                                                                                                                  \
        this->ParallelismType##Test(3);                                                                                \
    }                                                                                                                  \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##SweepBatchSizes)                                                 \
    {                                                                                                                  \
        this->mIsLongTest = true;                                                                                      \
        for (int batch_size : {2, 8, 15, 64, 73, 256})                                                                 \
        {                                                                                                              \
            this->ParallelismType##Test(1, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);                                  \
            this->ParallelismType##Test(2, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);                                  \
            this->ParallelismType##Test(3, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);                                  \
        }                                                                                                              \
    }                                                                                                                  \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##SweepBatchSizesGeglu)                                            \
    {                                                                                                                  \
        this->mIsLongTest = true;                                                                                      \
        this->mActType = tensorrt_llm::ActivationType::Geglu;                                                          \
        for (int batch_size : {2, 8, 15, 64, 73, 256})                                                                 \
        {                                                                                                              \
            this->ParallelismType##Test(1, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);                                  \
            this->ParallelismType##Test(2, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);                                  \
            this->ParallelismType##Test(3, this->DEFAULT_HIDDEN_SIZE, 4, batch_size);                                  \
        }                                                                                                              \
    }                                                                                                                  \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##NoBias)                                                          \
    {                                                                                                                  \
        this->mUseBias = false;                                                                                        \
        this->ParallelismType##Test();                                                                                 \
        this->ParallelismType##Test(2);                                                                                \
        this->ParallelismType##Test(3);                                                                                \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##Renorm)                                                          \
    {                                                                                                                  \
        this->mNormMode = MOEExpertScaleNormalizationMode::RENORMALIZE;                                                \
        this->ParallelismType##Test();                                                                                 \
        this->ParallelismType##Test(2);                                                                                \
        this->ParallelismType##Test(3);                                                                                \
    }                                                                                                                  \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##SparseMixer)                                                     \
    {                                                                                                                  \
        this->mNormMode = MOEExpertScaleNormalizationMode::SPARSE_MIXER;                                               \
        this->ParallelismType##Test();                                                                                 \
        this->ParallelismType##Test(2);                                                                                \
        /* k=3 is not supported for sparse mixer tests */                                                              \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##Gelu)                                                            \
    {                                                                                                                  \
        this->mActType = tensorrt_llm::ActivationType::Gelu;                                                           \
        this->ParallelismType##Test();                                                                                 \
        this->ParallelismType##Test(2);                                                                                \
        this->ParallelismType##Test(3);                                                                                \
    }                                                                                                                  \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##Silu)                                                            \
    {                                                                                                                  \
        this->mActType = tensorrt_llm::ActivationType::Silu;                                                           \
        this->ParallelismType##Test();                                                                                 \
        this->ParallelismType##Test(2);                                                                                \
        this->ParallelismType##Test(3);                                                                                \
    }                                                                                                                  \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##Geglu)                                                           \
    {                                                                                                                  \
        this->mActType = tensorrt_llm::ActivationType::Geglu;                                                          \
        this->ParallelismType##Test();                                                                                 \
        this->ParallelismType##Test(2);                                                                                \
        this->ParallelismType##Test(3);                                                                                \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##Swiglu)                                                          \
    {                                                                                                                  \
        this->mActType = tensorrt_llm::ActivationType::Swiglu;                                                         \
        this->ParallelismType##Test();                                                                                 \
        this->ParallelismType##Test(2);                                                                                \
        this->ParallelismType##Test(3);                                                                                \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##Mixtral8x7b)                                                     \
    {                                                                                                                  \
        this->mIsLongTest = true;                                                                                      \
        this->mUseBias = false;                                                                                        \
        this->mActType = tensorrt_llm::ActivationType::Swiglu;                                                         \
        this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;                         \
        this->ParallelismType##Test(2, 4096, 8);                                                                       \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##NonPowerOfTwo)                                                   \
    {                                                                                                                  \
        this->ParallelismType##Test(1, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
        this->ParallelismType##Test(2, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
        this->ParallelismType##Test(3, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##NonPowerOfTwoRenorm)                                             \
    {                                                                                                                  \
        this->mNormMode = tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE;                         \
        this->ParallelismType##Test(1, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
        this->ParallelismType##Test(2, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
        this->ParallelismType##Test(3, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##NonPowerOfTwoSwiglu)                                             \
    {                                                                                                                  \
        this->mActType = tensorrt_llm::ActivationType::Swiglu;                                                         \
        this->ParallelismType##Test(1, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
        this->ParallelismType##Test(2, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
        this->ParallelismType##Test(3, this->DEFAULT_HIDDEN_SIZE, 10);                                                 \
    }                                                                                                                  \
                                                                                                                       \
    TYPED_TEST(MixtureOfExpertsTest, ParallelismType##ManyExperts)                                                     \
    {                                                                                                                  \
        this->mIsLongTest = true;                                                                                      \
        /* This test is very slow. Only do one k value */                                                              \
        this->ParallelismType##Test(2, this->MINIMUM_ALIGNMENT, 512);                                                  \
    }

PARALLEL_TEST_SUITE(ExpertParallel)
PARALLEL_TEST_SUITE(TensorParallel)
PARALLEL_TEST_SUITE(MixedParallel)

TYPED_TEST(MixtureOfExpertsTest, ConfigSweep)
{
    this->mIsLongTest = true;
    auto genConfigName = [](auto conf) -> std::string
    {
        using namespace tensorrt_llm::cutlass_extensions;
        std::stringstream tactic;
        tactic << "sm" << conf.sm_version << " tactic with tile shape ";
        if (conf.is_tma_warp_specialized)
        {
            tactic << conf.getTileConfigAsInt() << " and cluster shape " << (int) conf.cluster_shape
                   << " mainloop sched " << (int) conf.mainloop_schedule << " epi sched "
                   << (int) conf.epilogue_schedule;
        }
        else if (conf.tile_config_sm80 != CutlassTileConfig::ChooseWithHeuristic)
        {
            tactic << (int) conf.getTileConfigAsInt() << " and stages " << (int) conf.stages << " split k "
                   << (int) conf.split_k_factor;
        }
        else
        {
            return {};
        }
        return tactic.str();
    };

    auto activation_pool = std::vector{
        tensorrt_llm::ActivationType::Relu, tensorrt_llm::ActivationType::Swiglu, tensorrt_llm::ActivationType::Geglu};
    if (this->FP4)
        activation_pool = {tensorrt_llm::ActivationType::Relu};
    auto configs = this->getFilteredConfigs(getSMVersion());
    for (auto const activation_type : activation_pool)
    {
        for (auto conf1 : configs)
        {
            for (auto conf2 : configs)
            {
                auto name1 = genConfigName(conf1);
                auto name2 = genConfigName(conf2);
                if (name1.empty() || name2.empty())
                {
                    FAIL() << "Uninitialised tactic encountered";
                }
                ASSERT_NO_THROW({
                    this->mActType = activation_type;
                    for (int k = 1; k <= 3; k++)
                    {

                        this->mOverrideSelectedConfig1 = conf1;
                        this->mOverrideSelectedConfig2 = conf2;
                        this->BasicPermuteTest(k);
                        if (::testing::Test::HasFailure()) // Throw on test failure so we get the print message
                            throw std::runtime_error("Test k=" + std::to_string(k) + " Failed");
                    }
                }) << "Failed\nTactic 1: "
                   << name1 << "\nTactic 2: " << name2 << " and activation type: " << static_cast<int>(activation_type);
            }
        }
    }
}

TYPED_TEST(LargeMixtureOfExpertsTest, PermuteVeryLargeExperts)
{
    this->mIsLongTest = true;

    // Chosen so that hidden_size * inter_size * num_experts >> 2^32, but we can still fit in 80GB for `half`
    // Uses a non-power of two so any integer overflow will have bad alignment
    int64_t hidden_size = 31 * 1024;
    ASSERT_GT(hidden_size * hidden_size * 4, (int64_t) std::numeric_limits<int>::max() + 1ull);

    int64_t k = 2; // Use k=2 so all experts get a value
    // 3 tokens 4 experts are the defaults for BasicPermuteTest
    if (!this->checkSufficientTestMemory(3, hidden_size, 4, k))
    {
        GTEST_SKIP() << "Insufficient free memory for test";
    }

    this->BasicPermuteTest(k, hidden_size); // 4 x 32k x 128K experts
}

TYPED_TEST(LargeMixtureOfExpertsTest, PermuteVeryLongSequence)
{
    this->mIsLongTest = true;
    this->mUseBias = !this->FP8;

    using DataType = typename TypeParam::DataType;
    // Sequence * hidden size > INT32_MAX
    int64_t hidden_size = 2048ll;
    int64_t num_experts = 4;
    int64_t k = 1;
    int64_t tokens_to_test = 100;
    int64_t num_tokens = 2ull * 1024ll * 1024ll + tokens_to_test + 1ll;
    ASSERT_GT(hidden_size * (num_tokens - tokens_to_test), (uint64_t) std::numeric_limits<uint32_t>::max() + 1ull);

    if (!this->checkSufficientTestMemory(num_tokens, hidden_size, num_experts, k))
    {
        GTEST_SKIP() << "Insufficient free memory for test";
    }

    std::vector<DataType> hidden_states(hidden_size * num_tokens);
    this->mMaxInput = 1.f; // Any arbitrary non-zero value

    // All tokens to expert 0, so we catch the case where an expert has more than 2^32 tokens
    auto probs = this->repeat(std::vector<float>{1.f, 0.5f, 0.f, 0.f}, num_tokens);
    // Override the first few tokens to go to different experts.
    // This covers the regression case where an overflow only impacts one of the last experts
    // In particular the case when there are more than 2^32 elements before the last expert
    for (int i = 1; i < tokens_to_test; i++)
    {
        probs[i * num_experts + i % num_experts] = 2.f;
    }

    this->runMoEPermute({hidden_states}, {probs}, hidden_size, num_experts, k);

    // Just look at the first few tokens
    this->mTotalTokens = tokens_to_test;

    probs.resize(num_experts * this->mTotalTokens);
    hidden_states.resize(hidden_size * this->mTotalTokens);

    auto selected_expert = this->getDataFromDevice(this->mSelectedExpert, k * this->mTotalTokens);
    // We set the first few tokens to go to the corresponding i-th expert
    for (int i = 0; i < tokens_to_test; i++)
    {
        ASSERT_EQ(selected_expert[i], i % num_experts);
    }

    this->compareSoftmax(selected_expert, {probs});
    // Create a default vector for the reference outputs of the correct type for FP8
    std::vector<typename TypeParam::OutputType> unquant_states(this->mTotalTokens * hidden_size);
    this->compareFinal(selected_expert, {probs}, {unquant_states});
}

using MixtureOfExpertsProfilerTest = MixtureOfExpertsTest<WeightParams<half, half>>;

TEST_F(MixtureOfExpertsProfilerTest, TestGeneratedProfilerDistribution)
{
    //    int64_t num_tokens = 128;
    int64_t num_experts = 8;
    int64_t k = 2;

    GemmProfilerBackend backend;

    // We need to test different EP values to ensure the tokens are properly assigned
    for (int64_t num_tokens : {1, 128})
    {
        int64_t expanded_num_tokens = num_tokens * k;
        for (int ep : {1, 4, 8})
        {
            backend.init(this->mMoERunner, GemmProfilerBackend::GemmToProfile::GEMM_1, nvinfer1::DataType::kHALF,
                nvinfer1::DataType::kHALF, nvinfer1::DataType::kHALF, num_experts, k, 1024, 4096, {}, false, mUseLora,
                MOEParallelismConfig{1, 0, ep, ep - 1});

            auto ws_size = backend.getWorkspaceSize(num_tokens);
            auto workspace = this->allocBuffer<char>(ws_size);

            int64_t num_experts_per_node = num_experts / ep;

            backend.prepare(num_tokens, workspace, mStream->get());

            auto getNext = backend.getWorkspacePointerGenerator(workspace, num_tokens, getSMVersion() >= 90);
            auto const* expert_first_token_offset_size = reinterpret_cast<int64_t*>(getNext());
            auto const* source_to_dest_map = reinterpret_cast<int*>(getNext());
            auto const* dest_to_source_map = reinterpret_cast<int*>(getNext());
            auto const* token_selected_experts = reinterpret_cast<int*>(getNext());

            for (int sample = 0; sample < backend.NUM_ROUTING_SAMPLES; sample++)
            {
                auto host_expert_first_token_offset_size = getDataFromDevice(
                    expert_first_token_offset_size + sample * (num_experts_per_node + 1), num_experts_per_node + 1);
                auto host_source_to_dest_map
                    = getDataFromDevice(source_to_dest_map + sample * expanded_num_tokens, expanded_num_tokens);
                auto host_dest_to_source_map
                    = getDataFromDevice(dest_to_source_map + sample * expanded_num_tokens, expanded_num_tokens);
                auto host_token_selected_experts
                    = getDataFromDevice(token_selected_experts + sample * expanded_num_tokens, expanded_num_tokens);

                std::vector<int64_t> calculated_routing_values(num_experts_per_node + 1, 0);
                int skipped = 0;
                for (auto v : host_token_selected_experts)
                {
                    ASSERT_TRUE(v < num_experts_per_node || (v == num_experts && ep > 1));
                    skipped += (v == num_experts);
                    if (v < num_experts_per_node)
                    {
                        calculated_routing_values[v]++;
                    }
                }

                if (num_tokens > 1)
                {
                    // Check tokens are distributed between all EP ranks
                    // Statistically possible, but so unlikely that it should be considered a bug
                    ASSERT_TRUE(ep == 1 || skipped > 0);
                    // Check all experts get some tokens
                    ASSERT_EQ(std::count(calculated_routing_values.begin(), calculated_routing_values.end() - 1, 0), 0);

                    float p = 1.f / num_experts;
                    float variance = expanded_num_tokens * p * (1 - p);
                    float stddev = sqrt(variance);
                    float mean = expanded_num_tokens * p;
                    for (int i = 0; i < num_experts_per_node; i++)
                    {
                        // All values should be within three standard deviations of the mean
                        // 99.7% of values should fall within this range.
                        // We have NUM_ROUTING_SAMPLES * (8 + 2 + 1) = 176 cases so this is unlikely
                        // If the test changes to have a much larger number of cases this will need revisited
                        EXPECT_LE(abs(calculated_routing_values[i] - mean), 3 * stddev)
                            << "Expert " << i << " for sample " << sample << " has unbalanced token count "
                            << calculated_routing_values[i] << " vs mean value " << mean << " with standard deviation "
                            << stddev;
                    }
                }
                ASSERT_EQ(host_expert_first_token_offset_size.back(), expanded_num_tokens - skipped);

                std::exclusive_scan(calculated_routing_values.begin(), calculated_routing_values.end(),
                    calculated_routing_values.begin(), 0);
                ASSERT_TRUE(std::equal(calculated_routing_values.begin(), calculated_routing_values.end(),
                    host_expert_first_token_offset_size.begin()));

                std::fill(calculated_routing_values.begin(), calculated_routing_values.end(), 0);
                for (int64_t token_idx = 0; token_idx < num_tokens; token_idx++)
                {
                    for (int64_t k_idx = 0; k_idx < k; k_idx++)
                    {
                        int64_t idx = token_idx * k + k_idx;
                        int64_t expert_idx = host_token_selected_experts[idx];

                        if (expert_idx < num_experts)
                        {
                            int64_t source_location = k_idx * num_tokens + token_idx;
                            int64_t dest_location = host_expert_first_token_offset_size[expert_idx]
                                + calculated_routing_values[expert_idx];

                            ASSERT_EQ(host_source_to_dest_map[source_location], dest_location);
                            ASSERT_EQ(host_dest_to_source_map[dest_location], source_location);

                            calculated_routing_values[expert_idx]++;
                        }
                    }
                }
            }
        }
    }
}

using MixtureOfExpertsUnitTests = MixtureOfExpertsTest<WeightParams<half, half>>;

TEST_F(MixtureOfExpertsUnitTests, SparseMixerReferenceTest)
{
    // Test the sparse mixer reference implementation is doing the correct thing
    // This makes sure we are testing the correct behaviour
    this->mNumExperts = 4;
    this->mK = 2;
    auto res = this->sparseMixer({1.0f, 1.0f, -INFINITY, -INFINITY}, 0, 0, 0);
    ASSERT_FLOAT_EQ(res, 0.5f);
    res = this->sparseMixer({1.0f, 1.0f, -INFINITY, -INFINITY}, 0, 1, 1);
    ASSERT_FLOAT_EQ(res, 1.0f);

    res = this->sparseMixer({2.0f, 0.0f, -INFINITY, -INFINITY}, 0, 0, 0);
    ASSERT_FLOAT_EQ(res, 1.0f);
    res = this->sparseMixer({2.0f, 0.0f, -INFINITY, -INFINITY}, 0, 1, 1);
    ASSERT_FLOAT_EQ(res, 1.0f);

    res = this->sparseMixer({0.0f, 2.0f, -INFINITY, -INFINITY}, 0, 0, 1);
    ASSERT_FLOAT_EQ(res, 1.0f);
    res = this->sparseMixer({0.0f, 2.0f, -INFINITY, -INFINITY}, 0, 1, 0);
    ASSERT_FLOAT_EQ(res, 1.0f);

    res = this->sparseMixer({1.0f, 1.0f, 1.0f, -INFINITY}, 0, 0, 0);
    ASSERT_FLOAT_EQ(res, 1.f / 3.f);
    res = this->sparseMixer({1.0f, 1.0f, 1.0f, -INFINITY}, 0, 1, 1);
    ASSERT_FLOAT_EQ(res, 0.5f);
}
