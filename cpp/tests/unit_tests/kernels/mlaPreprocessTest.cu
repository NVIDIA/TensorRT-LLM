#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <algorithm>
#include <cstdint>
#include <gtest/gtest.h>

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/quantization.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"
#include "tensorrt_llm/runtime/bufferManager.h"

#include "tensorrt_llm/kernels/mlaKernels.h"
#include <random>

namespace
{

// copy matched kv cache data to kv_output
// kv_output {total_cached_token, num_head = 1, head_size(lora_size + rope_size)}
// compressed_kv_cache {batch, 1 (ignore v), max_seq_len / tokens_per_block, num_head, tokens_per_block, (lora_size +
// rope_size)}
template <typename T>
void loadPagedKvKernelRef(T* kv_output, tensorrt_llm::kernels::KVBlockArray const& compressed_kv_cache,
    int num_contexts, int64_t const* cu_ctx_cached_kv_lens, int head_dim)
{

    for (int b = 0; b < num_contexts; b++)
    {
        int const global_token_offset = cu_ctx_cached_kv_lens[b];
        int const current_token_len = cu_ctx_cached_kv_lens[b + 1] - cu_ctx_cached_kv_lens[b];
        for (int s = 0; s < current_token_len; s++)
        {
            for (int d = 0; d < head_dim; d++)
            {
                auto const* kv_src = reinterpret_cast<T const*>(compressed_kv_cache.getKBlockPtr(b, s));
                auto kv_block_idx = compressed_kv_cache.getKVLocalIdx(s, 0, head_dim, d);

                int const global_token_idx = global_token_offset + s;
                int const dst_idx = global_token_idx * head_dim + d;
                kv_output[dst_idx] = kv_src[kv_block_idx];
            }
        }
    }
}

// k {total_token, h, uncompressed_h=128}, v {total_token, h, uncompressed_h}, k_pe {total_token, h=1, rope_h}
// output {b, 2, ceil(max_seq / kv_cache_tokens_per_block), h, kv_cache_tokens_per_block, (uncompressed_h + rope_h)}
// copy k, v, k_pe to a continuous memory space (then it will be packed to kv_cache)
template <typename T>
void setPagedKvCacheForMLAKernelRef(T* output, T* const k_ptr, T* const v_ptr, T* const k_pe_ptr, int num_requests,
    int64_t const* cu_seq_lens, int const max_input_seq_len, int num_heads, int uncompressed_head_size, int rope_size,
    int kv_cache_tokens_per_block)
{
    int const kv_cache_size_per_block = num_heads * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size);
    int const kv_cache_block_num_per_seq
        = (max_input_seq_len + kv_cache_tokens_per_block - 1) / kv_cache_tokens_per_block;
    for (int b = 0; b < num_requests; b++)
    {
        int const global_token_offset = cu_seq_lens[b];
        int const current_token_len = cu_seq_lens[b + 1] - cu_seq_lens[b];
        for (int s = 0; s < current_token_len; s++)
        {
            int const global_token_idx = global_token_offset + s;
            int const kv_cache_block_offset_for_k
                = ((b * 2 * kv_cache_block_num_per_seq) + (s / kv_cache_tokens_per_block)) * kv_cache_size_per_block;
            int const kv_cache_block_offset_for_v
                = kv_cache_block_offset_for_k + (kv_cache_block_num_per_seq * kv_cache_size_per_block);
            for (int h = 0; h < num_heads; h++)
            {
                // copy k, v
                int const ld_kv_head_offset
                    = (global_token_idx * num_heads * uncompressed_head_size) + (h * uncompressed_head_size);
                int const ld_k_pe_head_offset = (global_token_idx * rope_size);
                for (int d = 0; d < uncompressed_head_size; d++)
                {
                    int const ld_kv_idx = ld_kv_head_offset + d;
                    int const st_k_idx = kv_cache_block_offset_for_k
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d;
                    int const st_v_idx = kv_cache_block_offset_for_v
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d;
                    output[st_k_idx] = k_ptr[ld_kv_idx];
                    output[st_v_idx] = v_ptr[ld_kv_idx];
                }
                // copy k_pe, head_num = 1
                for (int d = 0; d < rope_size; d++)
                {
                    int const ld_k_pe_idx = ld_k_pe_head_offset + d;
                    int const st_k_pe_idx = kv_cache_block_offset_for_k
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d
                        + uncompressed_head_size;
                    output[st_k_pe_idx] = k_pe_ptr[ld_k_pe_idx];
                }
            }
        }
    }
}

// ck or cv {total_cached_token, h, uncompressed_h=128}, ck_pe {total_cached_token, h=1, rope_h}
// uk or uv {total_uncached_token, h, uncompressed_h}, uk_pe {total_uncached_token, h=1, rope_h}
// output {b, 2, ceil(max_seq / kv_cache_tokens_per_block), h, kv_cache_tokens_per_block, (uncompressed_h + rope_h)}
// copy k, v, k_pe to a continuous memory space (then it will be packed to kv_cache)
template <typename T>
void setPagedKvCacheForMLAKernelRefV2(T* output, T* const ck_ptr, T* const cv_ptr, T* const ck_pe_ptr, T* const nk_ptr,
    T* const nv_ptr, T* const nk_pe_ptr, int num_requests, int64_t const* cu_ctx_cached_kv_lens,
    int64_t const* cu_seq_lens, int const max_input_seq_len, int num_heads, int uncompressed_head_size, int rope_size,
    int kv_cache_tokens_per_block)
{
    int const kv_cache_size_per_block = num_heads * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size);
    int const kv_cache_block_num_per_seq
        = (max_input_seq_len + kv_cache_tokens_per_block - 1) / kv_cache_tokens_per_block;
    for (int b = 0; b < num_requests; b++)
    {
        int const global_cached_token_offset = cu_ctx_cached_kv_lens[b];
        int const global_unchached_token_offset = cu_seq_lens[b] - cu_ctx_cached_kv_lens[b];
        int const current_token_len = cu_seq_lens[b + 1] - cu_seq_lens[b];
        int const current_cached_token_len = cu_ctx_cached_kv_lens[b + 1] - cu_ctx_cached_kv_lens[b];
        // int const current_uncached_token_len = current_token_len - current_cached_token_len;

        for (int s = 0; s < current_token_len; s++)
        {
            bool const is_cached = (s < current_cached_token_len);
            int const global_token_idx = is_cached ? global_cached_token_offset + s
                                                   : global_unchached_token_offset + (s - current_cached_token_len);
            int const kv_cache_block_offset_for_k
                = ((b * 2 * kv_cache_block_num_per_seq) + (s / kv_cache_tokens_per_block)) * kv_cache_size_per_block;
            int const kv_cache_block_offset_for_v
                = kv_cache_block_offset_for_k + (kv_cache_block_num_per_seq * kv_cache_size_per_block);
            auto const k_ptr = is_cached ? ck_ptr : nk_ptr;
            auto const v_ptr = is_cached ? cv_ptr : nv_ptr;
            auto const k_pe_ptr = is_cached ? ck_pe_ptr : nk_pe_ptr;
            for (int h = 0; h < num_heads; h++)
            {
                // copy k, v
                int const ld_kv_head_offset
                    = (global_token_idx * num_heads * uncompressed_head_size) + (h * uncompressed_head_size);
                int const ld_k_pe_head_offset = (global_token_idx * rope_size);
                for (int d = 0; d < uncompressed_head_size; d++)
                {
                    int const ld_kv_idx = ld_kv_head_offset + d;
                    int const st_k_idx = kv_cache_block_offset_for_k
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d;
                    int const st_v_idx = kv_cache_block_offset_for_v
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d;
                    output[st_k_idx] = k_ptr[ld_kv_idx];
                    output[st_v_idx] = v_ptr[ld_kv_idx];
                }
                // copy k_pe, head_num = 1
                for (int d = 0; d < rope_size; d++)
                {
                    int const ld_k_pe_idx = ld_k_pe_head_offset + d;
                    int const st_k_pe_idx = kv_cache_block_offset_for_k
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d
                        + uncompressed_head_size;
                    output[st_k_pe_idx] = k_pe_ptr[ld_k_pe_idx];
                }
            }
        }
    }
}

// compressed_kv_cache {batch, 1 (ignore v), max_seq_len / tokens_per_block, num_head=1, tokens_per_block, (lora_size +
// rope_size)}
// kv {total_uncached_tokens, h_k=1, lora_d}, k_pe {total_uncached_tokens, h_kpe=128, rope_d}
template <typename T>
void appendPagedKvForMLAKernelRef(tensorrt_llm::kernels::KVBlockArray& kv_cache, T* const compressed_kv_ptr,
    T* const k_pe_ptr, int const num_requests, int64_t const* cu_ctx_cached_kv_lens, int64_t const* cu_seq_lens,
    int k_pe_head_num, int lora_size, int rope_size)
{
    assert(k_pe_head_num == 1);
    for (int b = 0; b < num_requests; b++)
    {
        int const global_token_offset = cu_seq_lens[b] - cu_ctx_cached_kv_lens[b];
        int const cached_kv_len = cu_ctx_cached_kv_lens[b + 1] - cu_ctx_cached_kv_lens[b];
        int const uncached_token_len = cu_seq_lens[b + 1] - cu_seq_lens[b] - cached_kv_len;
        for (int s = 0; s < uncached_token_len; s++)
        {
            int const ld_kv_offset = (global_token_offset + s) * lora_size;
            int const ld_k_pe_offset = (global_token_offset + s) * k_pe_head_num * rope_size;
            auto* kv_cache_ptr = reinterpret_cast<T*>(kv_cache.getKBlockPtr(b, cached_kv_len + s));
            // copy kv
            for (int d = 0; d < lora_size; d++)
            {
                int const ld_kv_idx = ld_kv_offset + d;
                int const kv_cache_idx_in_block
                    = kv_cache.getKVLocalIdx(cached_kv_len + s, 0, lora_size + rope_size, d);
                kv_cache_ptr[kv_cache_idx_in_block] = compressed_kv_ptr[ld_kv_idx];
            }
            // copy k_pe (we only copy the first head)
            for (int d = 0; d < rope_size; d++)
            {
                int const ld_k_pe_idx = ld_k_pe_offset + d;
                int const kv_cache_idx_in_block
                    = kv_cache.getKVLocalIdx(cached_kv_len + s, 0, lora_size + rope_size, d + lora_size);
                kv_cache_ptr[kv_cache_idx_in_block] = k_pe_ptr[ld_k_pe_idx];
            }
        }
    }
}

inline bool almostEqual(float a, float b, float atol = 1e-2, float rtol = 1e-3)
{
    if (isnan(a) || isnan(b))
    {
        return false;
    }
    return fabs(a - b) <= (atol + rtol * fabs(b));
}

} // namespace

template <typename _DataType>
class MlaPreprocessTest : public testing::Test
{
protected:
    using DataType = _DataType;
    std::shared_ptr<tensorrt_llm::runtime::BufferManager> mBufferManager;
    std::shared_ptr<tensorrt_llm::runtime::CudaStream> mStream;
    // kv_cache shape {batch, 2(k or v), max_seq_len / tokens_per_block, num_head, tokens_per_block, head_size}
    // k, v, k_pe shape {total_token, num_head, head_size(lora_size or rope_size, or uncompressed_head_size)}
    // offset shape {batch, 2, max_seq_len / tokens_per_block}
    // for KVBlockArray, we only allocate primary pool.
    // you can infer the allocateBuffers function for more details.
    tensorrt_llm::runtime::BufferManager::ITensorPtr h_kv_cache_tensor{nullptr}, h_kv_cache_tensor_ref{nullptr},
        d_kv_cache_tensor{nullptr}, d_compressed_kv_cache_tensor{nullptr}, d_compressed_kv_cache_tensor_ref{nullptr},
        h_compressed_kv_cache_tensor{nullptr}, h_compressed_kv_cache_tensor_ref{nullptr}, d_offset_tensor{nullptr},
        d_compressed_offset_tensor{nullptr}, d_cu_ctx_cached_kv_lens{nullptr}, d_cu_seq_lens{nullptr},
        h_offset_tensor{nullptr}, h_compressed_offset_tensor{nullptr}, h_cu_ctx_cached_kv_lens{nullptr},
        h_cu_seq_lens{nullptr},
        // for kernel 1
        d_kv_k_pe_tensor{nullptr}, h_kv_k_pe_tensor{nullptr}, h_kv_k_pe_tensor_ref{nullptr},
        // for kernel 2
        d_k_tensor{nullptr}, d_v_tensor{nullptr}, d_k_pe_tensor{nullptr}, h_k_tensor{nullptr}, h_v_tensor{nullptr},
        h_k_pe_tensor{nullptr},
        // for kernel 2 (new)
        d_k_tensor_cached{nullptr}, d_v_tensor_cached{nullptr}, d_k_pe_tensor_cached{nullptr},
        d_k_tensor_uncached{nullptr}, d_v_tensor_uncached{nullptr}, d_k_pe_tensor_uncached{nullptr},
        h_k_tensor_cached{nullptr}, h_v_tensor_cached{nullptr}, h_k_pe_tensor_cached{nullptr},
        h_k_tensor_uncached{nullptr}, h_v_tensor_uncached{nullptr}, h_k_pe_tensor_uncached{nullptr},
        // for kernel 3
        d_compressed_kv_tensor{nullptr}, d_k_pe_full_head_tensor{nullptr}, h_compressed_kv_tensor{nullptr},
        h_k_pe_full_head_tensor{nullptr};

    int mNumRequests{};
    int mMaxSeqLen{};
    int mMaxCachedSeqLen{};
    int mMaxUncachedSeqLen{};
    int mMaxBlockPerSeq{};
    int mTokensPerBlock{};
    int mNumHeadsCompressed{};
    int mNumHeadsUncompressed{};
    int mTotalTokens{};
    int mTotalCachedTokens{};
    int mTotalUncachedTokens{};
    int mLoraSize{};
    int mRopeSize{};
    int mUncompressedHeadSize{};

    std::mt19937 gen;

    void SetUp() override
    {
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping mla preprocess test";
        }
        mStream = std::make_shared<tensorrt_llm::runtime::CudaStream>();
        mBufferManager = std::make_shared<tensorrt_llm::runtime::BufferManager>(mStream);
        gen.seed(42U);
    }

    static bool shouldSkip()
    {
        return false;
    }

    void setDefaultParams()
    {
        this->mTokensPerBlock = 64;
        this->mNumHeadsCompressed = 1;
        this->mNumHeadsUncompressed = 128;
        this->mLoraSize = 512;
        this->mRopeSize = 64;
        this->mUncompressedHeadSize = 128;
        this->mMaxSeqLen = 0;
        this->mMaxCachedSeqLen = 0;
        this->mMaxUncachedSeqLen = 0;
    }

    template <typename T>
    void fillKVOffsetData(T* arr, size_t size, bool use_both_kv = true, int max_block_per_seq = 0)
    {
        if (use_both_kv)
        {
            for (int i = 0; i < size; i++)
            {
                arr[i] = static_cast<T>(i);
            }
        }
        else
        {
            int temp_idx = 0;
            for (int i = 0; i < size; i++)
            {
                bool is_v = (((i / max_block_per_seq) % 2) == 1);
                if (is_v)
                {
                    arr[i] = static_cast<T>(0);
                }
                else
                {
                    arr[i] = static_cast<T>(temp_idx);
                    temp_idx++;
                }
            }
        }
    }

    template <typename T>
    void fillArrayDataWithMod(T* arr, size_t size)
    {
        for (int i = 0; i < size; i++)
        {
            arr[i] = static_cast<T>(i % 448);
        }
    }

    int generateRandomSizeSmallerThan(int a)
    {
        if (a <= 0)
        {
            return 0;
        }
        std::uniform_int_distribution<> distrib(0, a - 1);
        // Generate and return the random number
        return int{distrib(gen)};
    }

    template <typename T>
    void memsetZeroDevice(T* ptr, size_t size)
    {
        hipMemset(ptr, 0, size * sizeof(T));
    }

    template <typename T>
    void memsetZeroHost(T* ptr, size_t size)
    {
        std::memset(ptr, 0, size * sizeof(T));
    }

    bool allocateBuffers()
    {
        using tensorrt_llm::runtime::BufferManager;
        using tensorrt_llm::runtime::CudaStream;
        using tensorrt_llm::runtime::ITensor;
        using tensorrt_llm::runtime::bufferCast;

        auto dtype = nvinfer1::DataType::kHALF;
        if constexpr (std::is_same_v<DataType, float>)
        {
            dtype = nvinfer1::DataType::kFLOAT;
        }
        else if constexpr (std::is_same_v<DataType, half>)
        {
            dtype = nvinfer1::DataType::kHALF;
        }
        else if constexpr (std::is_same_v<DataType, __hip_bfloat16>)
        {
            dtype = nvinfer1::DataType::kBF16;
        }
        else
        {
            return false;
        }
        this->h_cu_seq_lens = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests + 1}), nvinfer1::DataType::kINT64);
        this->h_cu_ctx_cached_kv_lens = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests + 1}), nvinfer1::DataType::kINT64);
        this->d_cu_seq_lens = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests + 1}), nvinfer1::DataType::kINT64);
        this->d_cu_ctx_cached_kv_lens = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests + 1}), nvinfer1::DataType::kINT64);
        {
            // set random sequence length
            auto* cu_seq_lens_temp_ptr = bufferCast<int64_t>(*(this->h_cu_seq_lens));
            auto* cu_ctx_cached_kv_lens_temp_ptr = bufferCast<int64_t>(*(this->h_cu_ctx_cached_kv_lens));
            cu_seq_lens_temp_ptr[0] = 0;
            cu_ctx_cached_kv_lens_temp_ptr[0] = 0;
            for (int i = 1; i <= this->mNumRequests; i++)
            {
                int temp_seq_len = generateRandomSizeSmallerThan(512);
                if (temp_seq_len <= 0)
                {
                    temp_seq_len = 1; // at least 1 token
                }
                int cached_seq_len = generateRandomSizeSmallerThan(temp_seq_len);
                this->mMaxSeqLen = std::max(temp_seq_len, this->mMaxSeqLen);
                this->mMaxCachedSeqLen = std::max(cached_seq_len, this->mMaxCachedSeqLen);
                this->mMaxUncachedSeqLen = std::max(temp_seq_len - cached_seq_len, this->mMaxUncachedSeqLen);
                this->mTotalTokens += temp_seq_len;
                this->mTotalCachedTokens += cached_seq_len;
                this->mTotalUncachedTokens += temp_seq_len - cached_seq_len;
                cu_seq_lens_temp_ptr[i] = cu_seq_lens_temp_ptr[i - 1] + temp_seq_len;
                cu_ctx_cached_kv_lens_temp_ptr[i] = cu_ctx_cached_kv_lens_temp_ptr[i - 1] + cached_seq_len;
                // std::cout << "batch " << i << "seq len: " << temp_seq_len << ", cached len: " << cached_seq_len
                //           << ", uncached len: " << temp_seq_len - cached_seq_len << std::endl;
            }
            hipMemcpy(this->d_cu_seq_lens->data(), this->h_cu_seq_lens->data(), this->h_cu_seq_lens->getSizeInBytes(),
                hipMemcpyHostToDevice);
            hipMemcpy(this->d_cu_ctx_cached_kv_lens->data(), this->h_cu_ctx_cached_kv_lens->data(),
                this->h_cu_ctx_cached_kv_lens->getSizeInBytes(), hipMemcpyHostToDevice);
        }

        // malloc kv_cache
        this->mMaxBlockPerSeq = (this->mMaxSeqLen + this->mTokensPerBlock - 1) / this->mTokensPerBlock;
        this->h_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq, this->mNumHeadsUncompressed,
                this->mTokensPerBlock, this->mUncompressedHeadSize + this->mRopeSize}),
            dtype);
        this->h_kv_cache_tensor_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq, this->mNumHeadsUncompressed,
                this->mTokensPerBlock, this->mUncompressedHeadSize + this->mRopeSize}),
            dtype);
        this->h_compressed_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 1, this->mMaxBlockPerSeq, this->mNumHeadsCompressed,
                this->mTokensPerBlock, this->mLoraSize + this->mRopeSize}),
            dtype);
        this->h_compressed_kv_cache_tensor_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 1, this->mMaxBlockPerSeq, this->mNumHeadsCompressed,
                this->mTokensPerBlock, this->mLoraSize + this->mRopeSize}),
            dtype);
        this->h_offset_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq}), nvinfer1::DataType::kINT32);
        this->h_compressed_offset_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq}), nvinfer1::DataType::kINT32);
        this->d_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq, this->mNumHeadsUncompressed,
                this->mTokensPerBlock, this->mUncompressedHeadSize + this->mRopeSize}),
            dtype);
        this->d_compressed_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 1, this->mMaxBlockPerSeq, this->mNumHeadsCompressed,
                this->mTokensPerBlock, this->mLoraSize + this->mRopeSize}),
            dtype);
        this->d_compressed_kv_cache_tensor_ref = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 1, this->mMaxBlockPerSeq, this->mNumHeadsCompressed,
                this->mTokensPerBlock, this->mLoraSize + this->mRopeSize}),
            dtype);
        this->d_offset_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq}), nvinfer1::DataType::kINT32);
        this->d_compressed_offset_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq}), nvinfer1::DataType::kINT32);
        {
            auto* kv_cache_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor));
            auto* kv_cache_ref_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor_ref));
            auto* compressed_kv_cache_ptr = bufferCast<DataType>(*(this->h_compressed_kv_cache_tensor));
            auto* compressed_kv_cache_ref_ptr = bufferCast<DataType>(*(this->h_compressed_kv_cache_tensor_ref));
            auto* offset_ptr = bufferCast<int32_t>(*(this->h_offset_tensor));
            auto* compressed_offset_ptr = bufferCast<int32_t>(*(this->h_compressed_offset_tensor));
            fillArrayDataWithMod(compressed_kv_cache_ptr, this->h_compressed_kv_cache_tensor->getSize());
            fillArrayDataWithMod(compressed_kv_cache_ref_ptr, this->h_compressed_kv_cache_tensor_ref->getSize());
            memsetZeroHost<DataType>(kv_cache_ptr, this->h_kv_cache_tensor->getSize());
            memsetZeroHost<DataType>(kv_cache_ref_ptr, this->h_kv_cache_tensor_ref->getSize());
            hipMemcpy(this->d_kv_cache_tensor->data(), this->h_kv_cache_tensor->data(),
                this->h_kv_cache_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            // fillArrayDataWithMod(offset_ptr, this->offset_tensor->getSize());
            fillKVOffsetData(
                compressed_offset_ptr, this->h_compressed_offset_tensor->getSize(), false, this->mMaxBlockPerSeq);
            hipMemcpy(this->d_compressed_kv_cache_tensor->data(), this->h_compressed_kv_cache_tensor->data(),
                this->h_compressed_kv_cache_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_compressed_kv_cache_tensor_ref->data(), this->h_compressed_kv_cache_tensor_ref->data(),
                this->h_compressed_kv_cache_tensor_ref->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_compressed_offset_tensor->data(), this->h_compressed_offset_tensor->data(),
                this->h_compressed_offset_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_offset_tensor->data(), this->h_offset_tensor->data(),
                this->h_offset_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
        }

        // kv + k_pe for loadCompressedPagedKvKernel (kernel 1)
        // std::cout << "kv_cache_tensor size: {" << this->mTotalCachedTokens << ", 1, " << this->mLoraSize +
        // this->mRopeSize <<  "}" << std::endl;
        this->h_kv_k_pe_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, 1, this->mLoraSize + this->mRopeSize}), dtype);
        this->h_kv_k_pe_tensor_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, 1, this->mLoraSize + this->mRopeSize}), dtype);
        this->d_kv_k_pe_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalCachedTokens, 1, this->mLoraSize + this->mRopeSize}), dtype);
        {
            auto* kv_k_pe_ptr = bufferCast<DataType>(*(this->h_kv_k_pe_tensor));
            auto* kv_k_pe_ref_ptr = bufferCast<DataType>(*(this->h_kv_k_pe_tensor_ref));
            memsetZeroHost<DataType>(kv_k_pe_ptr, this->h_kv_k_pe_tensor->getSize());
            memsetZeroHost<DataType>(kv_k_pe_ref_ptr, this->h_kv_k_pe_tensor_ref->getSize());
            hipMemcpy(this->d_kv_k_pe_tensor->data(), this->h_kv_k_pe_tensor->data(),
                this->h_kv_k_pe_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
        }
        // k, v, k_pe for setPagedKvCacheForMLAKernel (kernel 2)
        this->h_k_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}), dtype);
        this->h_v_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}), dtype);
        this->h_k_pe_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsCompressed, this->mRopeSize}), dtype);
        this->d_k_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}), dtype);
        this->d_v_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}), dtype);
        this->d_k_pe_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsCompressed, this->mRopeSize}), dtype);
        {
            auto* k_ptr = bufferCast<DataType>(*(this->h_k_tensor));
            auto* v_ptr = bufferCast<DataType>(*(this->h_v_tensor));
            auto* k_pe_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor));
            fillArrayDataWithMod(k_ptr, this->h_k_tensor->getSize());
            fillArrayDataWithMod(v_ptr, this->h_v_tensor->getSize());
            fillArrayDataWithMod(k_pe_ptr, this->h_k_pe_tensor->getSize());
            hipMemcpy(this->d_k_tensor->data(), this->h_k_tensor->data(), this->h_k_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice);
            hipMemcpy(this->d_v_tensor->data(), this->h_v_tensor->data(), this->h_v_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice);
            hipMemcpy(this->d_k_pe_tensor->data(), this->h_k_pe_tensor->data(), this->h_k_pe_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice);
        }
        // ck, cv, ck_pe, uk, uc, uk_pe for setPagedKvCacheForMLAKernelV2 (kernel 2)
        this->h_k_tensor_cached = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}),
            dtype);
        this->h_v_tensor_cached = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}),
            dtype);
        this->h_k_pe_tensor_cached = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, this->mNumHeadsCompressed, this->mRopeSize}), dtype);
        this->h_k_tensor_uncached = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalUncachedTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}),
            dtype);
        this->h_v_tensor_uncached = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalUncachedTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}),
            dtype);
        this->h_k_pe_tensor_uncached = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalUncachedTokens, this->mNumHeadsCompressed, this->mRopeSize}), dtype);
        this->d_k_tensor_cached = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalCachedTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}),
            dtype);
        this->d_v_tensor_cached = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalCachedTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}),
            dtype);
        this->d_k_pe_tensor_cached = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalCachedTokens, this->mNumHeadsCompressed, this->mRopeSize}), dtype);
        this->d_k_tensor_uncached = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalUncachedTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}),
            dtype);
        this->d_v_tensor_uncached = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalUncachedTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}),
            dtype);
        this->d_k_pe_tensor_uncached = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalUncachedTokens, this->mNumHeadsCompressed, this->mRopeSize}), dtype);
        {
            auto* k_cached_ptr = bufferCast<DataType>(*(this->h_k_tensor_cached));
            auto* v_cached_ptr = bufferCast<DataType>(*(this->h_v_tensor_cached));
            auto* k_pe_cached_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor_cached));
            auto* k_uncached_ptr = bufferCast<DataType>(*(this->h_k_tensor_uncached));
            auto* v_uncached_ptr = bufferCast<DataType>(*(this->h_v_tensor_uncached));
            auto* k_pe_uncached_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor_uncached));
            fillArrayDataWithMod(k_cached_ptr, this->h_k_tensor_cached->getSize());
            fillArrayDataWithMod(v_cached_ptr, this->h_v_tensor_cached->getSize());
            fillArrayDataWithMod(k_pe_cached_ptr, this->h_k_pe_tensor_cached->getSize());
            fillArrayDataWithMod(k_uncached_ptr, this->h_k_tensor_uncached->getSize());
            fillArrayDataWithMod(v_uncached_ptr, this->h_v_tensor_uncached->getSize());
            fillArrayDataWithMod(k_pe_uncached_ptr, this->h_k_pe_tensor_uncached->getSize());
            hipMemcpy(this->d_k_tensor_cached->data(), this->h_k_tensor_cached->data(),
                this->h_k_tensor_cached->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_v_tensor_cached->data(), this->h_v_tensor_cached->data(),
                this->h_v_tensor_cached->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_k_pe_tensor_cached->data(), this->h_k_pe_tensor_cached->data(),
                this->h_k_pe_tensor_cached->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_k_tensor_uncached->data(), this->h_k_tensor_uncached->data(),
                this->h_k_tensor_uncached->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_v_tensor_uncached->data(), this->h_v_tensor_uncached->data(),
                this->h_v_tensor_uncached->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_k_pe_tensor_uncached->data(), this->h_k_pe_tensor_uncached->data(),
                this->h_k_pe_tensor_uncached->getSizeInBytes(), hipMemcpyHostToDevice);
        }
        // compressed_kv, k_pe_full_head for setCompressedPagedKvForMLAKernel (kernel 3)
        this->h_compressed_kv_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalUncachedTokens, 1, this->mLoraSize}), dtype);
        this->h_k_pe_full_head_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalUncachedTokens, 1, this->mRopeSize}), dtype);
        this->d_compressed_kv_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalUncachedTokens, 1, this->mLoraSize}), dtype);
        this->d_k_pe_full_head_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalUncachedTokens, 1, this->mRopeSize}), dtype);

        {
            auto* compressed_kv_ptr = bufferCast<DataType>(*(this->h_compressed_kv_tensor));
            auto* k_pe_full_head_ptr = bufferCast<DataType>(*(this->h_k_pe_full_head_tensor));
            fillArrayDataWithMod(compressed_kv_ptr, this->h_compressed_kv_tensor->getSize());
            fillArrayDataWithMod(k_pe_full_head_ptr, this->h_k_pe_full_head_tensor->getSize());
            hipMemcpy(this->d_compressed_kv_tensor->data(), this->h_compressed_kv_tensor->data(),
                this->h_compressed_kv_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_k_pe_full_head_tensor->data(), this->h_k_pe_full_head_tensor->data(),
                this->h_k_pe_full_head_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
        }
        return true;
    }

    void PerformLoadPagedKV()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* kv_k_pe_ptr = bufferCast<DataType>(*(this->d_kv_k_pe_tensor));
        auto* compressed_kv_cache_ptr = bufferCast<DataType>(*(this->d_compressed_kv_cache_tensor));
        auto* offset_ptr = bufferCast<int32_t>(*(this->d_compressed_offset_tensor));
        auto* cu_ctx_cached_kv_lens_ptr = bufferCast<int64_t>(*(this->d_cu_ctx_cached_kv_lens));
        tensorrt_llm::kernels::KVBlockArray kv_cache(this->mNumRequests, this->mMaxBlockPerSeq, this->mTokensPerBlock,
            sizeof(DataType) * 1 * (this->mLoraSize + this->mRopeSize), 0, 0, 0, 0, compressed_kv_cache_ptr, nullptr,
            reinterpret_cast<tensorrt_llm::kernels::KVBlockArrayForContextFMHA::DataType*>(offset_ptr));
        tensorrt_llm::kernels::invokeMLALoadPagedKV<DataType>(kv_k_pe_ptr, kv_cache, this->mNumRequests,
            cu_ctx_cached_kv_lens_ptr, this->mMaxCachedSeqLen, this->mLoraSize + this->mRopeSize, this->mStream->get());
        hipStreamSynchronize(this->mStream->get());
        hipMemcpy(this->h_kv_k_pe_tensor->data(), this->d_kv_k_pe_tensor->data(),
            this->d_kv_k_pe_tensor->getSizeInBytes(), hipMemcpyDeviceToHost);
    }

    void PerformLoadPagedKVRef()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* kv_k_pe_ptr = bufferCast<DataType>(*(this->h_kv_k_pe_tensor_ref));
        auto* compressed_kv_cache_ptr = bufferCast<DataType>(*(this->h_compressed_kv_cache_tensor));
        auto* offset_ptr = bufferCast<int32_t>(*(this->h_compressed_offset_tensor));
        auto* cu_ctx_cached_kv_lens_ptr = bufferCast<int64_t>(*(this->h_cu_ctx_cached_kv_lens));
        tensorrt_llm::kernels::KVBlockArray kv_cache(this->mNumRequests, this->mMaxBlockPerSeq, this->mTokensPerBlock,
            sizeof(DataType) * 1 * (this->mLoraSize + this->mRopeSize), 0, 0, 0, 0, compressed_kv_cache_ptr, nullptr,
            reinterpret_cast<tensorrt_llm::kernels::KVBlockArrayForContextFMHA::DataType*>(offset_ptr));
        loadPagedKvKernelRef(
            kv_k_pe_ptr, kv_cache, this->mNumRequests, cu_ctx_cached_kv_lens_ptr, this->mLoraSize + this->mRopeSize);
    }

    void PerformSetPagedKV()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* k_ptr = bufferCast<DataType>(*(this->d_k_tensor));
        auto* v_ptr = bufferCast<DataType>(*(this->d_v_tensor));
        auto* k_pe_ptr = bufferCast<DataType>(*(this->d_k_pe_tensor));
        auto* kv_cache_ptr = bufferCast<DataType>(*(this->d_kv_cache_tensor));
        auto* cu_seq_lens_ptr = bufferCast<int64_t>(*(this->d_cu_seq_lens));
        tensorrt_llm::kernels::invokeMLASetPagedKV<DataType>(kv_cache_ptr, k_ptr, v_ptr, k_pe_ptr, this->mNumRequests,
            cu_seq_lens_ptr, this->mMaxSeqLen, this->mNumHeadsUncompressed, this->mUncompressedHeadSize,
            this->mRopeSize, this->mTokensPerBlock, this->mStream->get());
        hipStreamSynchronize(this->mStream->get());
        hipMemcpy(this->h_kv_cache_tensor->data(), this->d_kv_cache_tensor->data(),
            this->d_kv_cache_tensor->getSizeInBytes(), hipMemcpyDeviceToHost);
    }

    void PerformSetPagedKVRef()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* k_ptr = bufferCast<DataType>(*(this->h_k_tensor));
        auto* v_ptr = bufferCast<DataType>(*(this->h_v_tensor));
        auto* k_pe_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor));
        auto* kv_cache_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor_ref));
        auto* cu_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_seq_lens));
        setPagedKvCacheForMLAKernelRef(kv_cache_ptr, k_ptr, v_ptr, k_pe_ptr, this->mNumRequests, cu_seq_lens_ptr,
            this->mMaxSeqLen, this->mNumHeadsUncompressed, this->mUncompressedHeadSize, this->mRopeSize,
            this->mTokensPerBlock);
    }

    void PerformSetPagedKVV2()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* k_cached_ptr = bufferCast<DataType>(*(this->d_k_tensor_cached));
        auto* v_cached_ptr = bufferCast<DataType>(*(this->d_v_tensor_cached));
        auto* k_pe_cached_ptr = bufferCast<DataType>(*(this->d_k_pe_tensor_cached));
        auto* k_uncached_ptr = bufferCast<DataType>(*(this->d_k_tensor_uncached));
        auto* v_uncached_ptr = bufferCast<DataType>(*(this->d_v_tensor_uncached));
        auto* k_pe_uncached_ptr = bufferCast<DataType>(*(this->d_k_pe_tensor_uncached));
        auto* cu_ctx_cached_kv_lens_ptr = bufferCast<int64_t>(*(this->d_cu_ctx_cached_kv_lens));
        auto* kv_cache_ptr = bufferCast<DataType>(*(this->d_kv_cache_tensor));
        auto* cu_seq_lens_ptr = bufferCast<int64_t>(*(this->d_cu_seq_lens));
        tensorrt_llm::kernels::invokeMLASetPagedKVV2<DataType>(kv_cache_ptr, k_cached_ptr, v_cached_ptr,
            k_pe_cached_ptr, k_uncached_ptr, v_uncached_ptr, k_pe_uncached_ptr, this->mNumRequests,
            cu_ctx_cached_kv_lens_ptr, cu_seq_lens_ptr, this->mMaxSeqLen, this->mNumHeadsUncompressed,
            this->mUncompressedHeadSize, this->mRopeSize, this->mTokensPerBlock, this->mStream->get());
        hipStreamSynchronize(this->mStream->get());
        hipMemcpy(this->h_kv_cache_tensor->data(), this->d_kv_cache_tensor->data(),
            this->d_kv_cache_tensor->getSizeInBytes(), hipMemcpyDeviceToHost);
    }

    void PerformSetPagedKVV2Ref()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* k_cached_ptr = bufferCast<DataType>(*(this->h_k_tensor_cached));
        auto* v_cached_ptr = bufferCast<DataType>(*(this->h_v_tensor_cached));
        auto* k_pe_cached_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor_cached));
        auto* k_uncached_ptr = bufferCast<DataType>(*(this->h_k_tensor_uncached));
        auto* v_uncached_ptr = bufferCast<DataType>(*(this->h_v_tensor_uncached));
        auto* k_pe_uncached_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor_uncached));
        auto* cu_ctx_cached_kv_lens_ptr = bufferCast<int64_t>(*(this->h_cu_ctx_cached_kv_lens));
        auto* kv_cache_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor_ref));
        auto* cu_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_seq_lens));
        setPagedKvCacheForMLAKernelRefV2(kv_cache_ptr, k_cached_ptr, v_cached_ptr, k_pe_cached_ptr, k_uncached_ptr,
            v_uncached_ptr, k_pe_uncached_ptr, this->mNumRequests, cu_ctx_cached_kv_lens_ptr, cu_seq_lens_ptr,
            this->mMaxSeqLen, this->mNumHeadsUncompressed, this->mUncompressedHeadSize, this->mRopeSize,
            this->mTokensPerBlock);
    }

    void PerformAppendPagedKV()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* compressed_kv_ptr = bufferCast<DataType>(*(this->d_compressed_kv_tensor));
        auto* k_pe_full_head_ptr = bufferCast<DataType>(*(this->d_k_pe_full_head_tensor));
        auto* offset_ptr = bufferCast<int32_t>(*(this->d_compressed_offset_tensor));
        auto* compressed_kv_cache_ptr = bufferCast<DataType>(*(this->d_compressed_kv_cache_tensor));
        auto* cu_ctx_cached_kv_lens_ptr = bufferCast<int64_t>(*(this->d_cu_ctx_cached_kv_lens));
        auto* cu_seq_lens_ptr = bufferCast<int64_t>(*(this->d_cu_seq_lens));
        tensorrt_llm::kernels::KVBlockArray kv_cache(this->mNumRequests, this->mMaxBlockPerSeq, this->mTokensPerBlock,
            sizeof(DataType) * 1 * (this->mLoraSize + this->mRopeSize), 0, 0, 0, 0, compressed_kv_cache_ptr, nullptr,
            reinterpret_cast<tensorrt_llm::kernels::KVBlockArrayForContextFMHA::DataType*>(offset_ptr));
        tensorrt_llm::kernels::invokeMLAAppendPagedKV<DataType>(kv_cache, compressed_kv_ptr, k_pe_full_head_ptr,
            this->mNumRequests, cu_ctx_cached_kv_lens_ptr, cu_seq_lens_ptr, this->mMaxUncachedSeqLen,
            this->mLoraSize + this->mRopeSize, this->mStream->get());
        hipStreamSynchronize(this->mStream->get());
        hipMemcpy(this->h_compressed_kv_cache_tensor->data(), this->d_compressed_kv_cache_tensor->data(),
            this->d_compressed_kv_cache_tensor->getSizeInBytes(), hipMemcpyDeviceToHost);
    }

    void PerformAppendPagedKVRef()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* compressed_kv_ptr = bufferCast<DataType>(*(this->h_compressed_kv_tensor));
        auto* k_pe_full_head_ptr = bufferCast<DataType>(*(this->h_k_pe_full_head_tensor));
        auto* offset_ptr = bufferCast<int32_t>(*(this->h_compressed_offset_tensor));
        auto* compressed_kv_cache_ptr = bufferCast<DataType>(*(this->h_compressed_kv_cache_tensor_ref));
        auto* cu_ctx_cached_kv_lens_ptr = bufferCast<int64_t>(*(this->h_cu_ctx_cached_kv_lens));
        auto* cu_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_seq_lens));
        tensorrt_llm::kernels::KVBlockArray kv_cache(this->mNumRequests, this->mMaxBlockPerSeq, this->mTokensPerBlock,
            sizeof(DataType) * 1 * (this->mLoraSize + this->mRopeSize), 0, 0, 0, 0, compressed_kv_cache_ptr, nullptr,
            reinterpret_cast<tensorrt_llm::kernels::KVBlockArrayForContextFMHA::DataType*>(offset_ptr));
        // currently k_pe_head_num = 1
        appendPagedKvForMLAKernelRef(kv_cache, compressed_kv_ptr, k_pe_full_head_ptr, this->mNumRequests,
            cu_ctx_cached_kv_lens_ptr, cu_seq_lens_ptr, 1, this->mLoraSize, this->mRopeSize);
    }

    template <typename T>
    bool CheckEqual(T const* expected, T const* output, size_t size)
    {
        for (int i = 0; i < size; i++)
        {
            if (!almostEqual(expected[i], output[i], 1e-3, 1e-3))
            {
                TLLM_LOG_ERROR("Mismatch input value. Position of inputs: %d, expected value: %f, output value: %f", i,
                    static_cast<float>(expected[i]), static_cast<float>(output[i]));
                return false;
            }
        }
        return true;
    }
};

using MLATypes = ::testing::Types<half, __hip_bfloat16, float>;
TYPED_TEST_SUITE(MlaPreprocessTest, MLATypes);

TYPED_TEST(MlaPreprocessTest, MLAPreprocessDefault)
{
    using tensorrt_llm::runtime::bufferCast;
    using DataType = typename TestFixture::DataType;
    this->mNumRequests = 8;
    this->setDefaultParams();
    this->allocateBuffers();

    sync_check_cuda_error(this->mStream->get());
    bool allEqual{true};

    this->PerformLoadPagedKV();
    sync_check_cuda_error(this->mStream->get());
    this->PerformLoadPagedKVRef();
    auto* kv_k_pe_ptr = bufferCast<DataType>(*(this->h_kv_k_pe_tensor));
    auto* kv_k_pe_ref_ptr = bufferCast<DataType>(*(this->h_kv_k_pe_tensor_ref));
    allEqual = this->CheckEqual(kv_k_pe_ref_ptr, kv_k_pe_ptr, this->h_kv_k_pe_tensor->getSize());
    EXPECT_TRUE(allEqual);

    this->PerformSetPagedKVV2();
    sync_check_cuda_error(this->mStream->get());
    this->PerformSetPagedKVV2Ref();
    auto* kv_cache_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor));
    auto* kv_cache_ref_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor_ref));
    allEqual = this->CheckEqual(kv_cache_ref_ptr, kv_cache_ptr, this->h_kv_cache_tensor->getSize());
    EXPECT_TRUE(allEqual);

    this->PerformAppendPagedKV();
    sync_check_cuda_error(this->mStream->get());
    this->PerformAppendPagedKVRef();
    auto* compressed_kv_cache_ptr = bufferCast<DataType>(*(this->h_compressed_kv_cache_tensor));
    auto* compressed_kv_cache_ref_ptr = bufferCast<DataType>(*(this->h_compressed_kv_cache_tensor_ref));
    allEqual = this->CheckEqual(
        compressed_kv_cache_ref_ptr, compressed_kv_cache_ptr, this->h_compressed_kv_cache_tensor->getSize());
    EXPECT_TRUE(allEqual);
}
