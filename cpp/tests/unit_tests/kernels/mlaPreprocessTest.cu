#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <algorithm>
#include <cstdint>
#include <gtest/gtest.h>

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/quantization.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"
#include "tensorrt_llm/runtime/bufferManager.h"

#include "tensorrt_llm/kernels/mlaKernels.h"
#include <random>

namespace
{

// copy matched kv cache data to compressed_kv_output and k_pe_output
// compressed_kv_output {total_cached_token, lora_size}
// k_pe_output {total_cached_token, rope_size}
// compressed_kv_cache {batch, 1 (ignore v), max_seq_len / tokens_per_block, num_head, tokens_per_block, (lora_size +
// rope_size)}
template <typename T, typename TCache>
void loadPagedKvKernelRef(T* compressed_kv_output, T* k_pe_output,
    tensorrt_llm::kernels::KVBlockArray const& compressed_kv_cache, int num_contexts,
    int64_t const* cu_ctx_cached_kv_lens, int const lora_size, int const rope_size,
    float const* kv_scale_quant_orig_ptr)
{
    static_assert(std::is_same_v<T, TCache> || std::is_same_v<TCache, __hip_fp8_e4m3_fnuz>,
        "TCache must be either the same type as T or __hip_fp8_e4m3_fnuz");
    int const head_dim = lora_size + rope_size;
    float const kv_scale_quant_orig = kv_scale_quant_orig_ptr ? kv_scale_quant_orig_ptr[0] : 1.0f;
    for (int b = 0; b < num_contexts; b++)
    {
        int const global_token_offset = cu_ctx_cached_kv_lens[b];
        int const current_token_len = cu_ctx_cached_kv_lens[b + 1] - cu_ctx_cached_kv_lens[b];
        for (int s = 0; s < current_token_len; s++)
        {
            int const global_token_idx = global_token_offset + s;
            for (int d = 0; d < head_dim; d++)
            {
                auto const* kv_src = reinterpret_cast<TCache const*>(compressed_kv_cache.getKBlockPtr(b, s));
                auto kv_block_idx = compressed_kv_cache.getKVLocalIdx(s, 0, head_dim, d);

                auto src_data = kv_src[kv_block_idx];
                T data;
                if constexpr (std::is_same_v<TCache, __hip_fp8_e4m3_fnuz>)
                {
                    data = T(float(src_data) * kv_scale_quant_orig);
                }
                else
                {
                    data = src_data;
                }
                if (d < lora_size)
                {
                    compressed_kv_output[global_token_idx * lora_size + d] = data;
                }
                else
                {
                    k_pe_output[global_token_idx * rope_size + (d - lora_size)] = data;
                }
            }
        }
    }
}

// k {total_token, h, uncompressed_h=128}, v {total_token, h, uncompressed_h}, k_pe {total_token, h=1, rope_h}
// output {b, 2, ceil(max_seq / kv_cache_tokens_per_block), h, kv_cache_tokens_per_block, (uncompressed_h + rope_h)}
// copy k, v, k_pe to a continuous memory space (then it will be packed to kv_cache)
template <typename T>
void setPagedKvCacheForMLAKernelRef(T* output, T* const k_ptr, T* const v_ptr, T* const k_pe_ptr, int num_requests,
    int64_t const* cu_seq_lens, int const max_input_seq_len, int num_heads, int uncompressed_head_size, int rope_size,
    int kv_cache_tokens_per_block, int64_t kv_token_stride)
{
    int const kv_cache_size_per_block = num_heads * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size);
    int const kv_cache_block_num_per_seq
        = (max_input_seq_len + kv_cache_tokens_per_block - 1) / kv_cache_tokens_per_block;
    for (int b = 0; b < num_requests; b++)
    {
        int const global_token_offset = cu_seq_lens[b];
        int const current_token_len = cu_seq_lens[b + 1] - cu_seq_lens[b];
        for (int s = 0; s < current_token_len; s++)
        {
            int const global_token_idx = global_token_offset + s;
            int const kv_cache_block_offset_for_k
                = ((b * 2 * kv_cache_block_num_per_seq) + (s / kv_cache_tokens_per_block)) * kv_cache_size_per_block;
            int const kv_cache_block_offset_for_v
                = kv_cache_block_offset_for_k + (kv_cache_block_num_per_seq * kv_cache_size_per_block);
            for (int h = 0; h < num_heads; h++)
            {
                // copy k, v
                int const ld_kv_head_offset = (global_token_idx * kv_token_stride) + (h * uncompressed_head_size);
                int const ld_k_pe_head_offset = (global_token_idx * rope_size);
                for (int d = 0; d < uncompressed_head_size; d++)
                {
                    int const ld_kv_idx = ld_kv_head_offset + d;
                    int const st_k_idx = kv_cache_block_offset_for_k
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d;
                    int const st_v_idx = kv_cache_block_offset_for_v
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d;
                    output[st_k_idx] = k_ptr[ld_kv_idx];
                    output[st_v_idx] = v_ptr[ld_kv_idx];
                }
                // copy k_pe, head_num = 1
                for (int d = 0; d < rope_size; d++)
                {
                    int const ld_k_pe_idx = ld_k_pe_head_offset + d;
                    int const st_k_pe_idx = kv_cache_block_offset_for_k
                        + h * kv_cache_tokens_per_block * (uncompressed_head_size + rope_size)
                        + (s % kv_cache_tokens_per_block) * (uncompressed_head_size + rope_size) + d
                        + uncompressed_head_size;
                    output[st_k_pe_idx] = k_pe_ptr[ld_k_pe_idx];
                }
            }
        }
    }
}

inline bool almostEqual(float a, float b, float atol = 1e-2, float rtol = 1e-3)
{
    if (isnan(a) || isnan(b))
    {
        return false;
    }
    return fabs(a - b) <= (atol + rtol * fabs(b));
}

} // namespace

template <typename Typepair>
class MlaPreprocessTest : public testing::Test
{
protected:
    using DataType = typename Typepair::first_type;
    using TCache = typename Typepair::second_type;
    static_assert(std::is_same_v<DataType, TCache> || std::is_same_v<TCache, __hip_fp8_e4m3_fnuz>,
        "TCache must be either the same type as DataType or __hip_fp8_e4m3_fnuz");
    std::shared_ptr<tensorrt_llm::runtime::BufferManager> mBufferManager;
    std::shared_ptr<tensorrt_llm::runtime::CudaStream> mStream;
    // kv_cache shape {batch, 2(k or v), max_seq_len / tokens_per_block, num_head, tokens_per_block, head_size}
    // k, v, k_pe shape {total_token, num_head, head_size(lora_size or rope_size, or uncompressed_head_size)}
    // offset shape {batch, 2, max_seq_len / tokens_per_block}
    // for KVBlockArray, we only allocate primary pool.
    // you can infer the allocateBuffers function for more details.
    tensorrt_llm::runtime::BufferManager::ITensorPtr h_kv_cache_tensor{nullptr}, h_kv_cache_tensor_ref{nullptr},
        d_kv_cache_tensor{nullptr}, d_compressed_kv_cache_tensor{nullptr}, d_compressed_kv_cache_tensor_ref{nullptr},
        h_compressed_kv_cache_tensor{nullptr}, h_compressed_kv_cache_tensor_ref{nullptr}, d_offset_tensor{nullptr},
        d_compressed_offset_tensor{nullptr}, d_cu_ctx_cached_kv_lens{nullptr}, d_cu_seq_lens{nullptr},
        h_offset_tensor{nullptr}, h_compressed_offset_tensor{nullptr}, h_cu_ctx_cached_kv_lens{nullptr},
        h_cu_seq_lens{nullptr}, h_kv_scale_orig_quant{nullptr}, d_kv_scale_orig_quant{nullptr},
        h_kv_scale_quant_orig{nullptr}, d_kv_scale_quant_orig{nullptr},
        // for kernel 1
        d_compressed_kv_output{nullptr}, h_compressed_kv_output{nullptr}, h_compressed_kv_output_ref{nullptr},
        d_k_pe_output{nullptr}, h_k_pe_output{nullptr}, h_k_pe_output_ref{nullptr},
        // for kernel 2
        d_k_tensor{nullptr}, d_v_tensor{nullptr}, d_k_pe_tensor{nullptr}, h_k_tensor{nullptr}, h_v_tensor{nullptr},
        h_k_pe_tensor{nullptr};

    int mNumRequests{};
    int mMaxSeqLen{};
    int mMaxCachedSeqLen{};
    int mMaxUncachedSeqLen{};
    int mMaxBlockPerSeq{};
    int mTokensPerBlock{};
    int mNumHeadsCompressed{};
    int mNumHeadsUncompressed{};
    int mTotalTokens{};
    int mTotalCachedTokens{};
    int mTotalUncachedTokens{};
    int mLoraSize{};
    int mRopeSize{};
    int mUncompressedHeadSize{};
    int64_t mKvTokenStride{};

    std::mt19937 gen;

    void SetUp() override
    {
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping mla preprocess test";
        }
        mStream = std::make_shared<tensorrt_llm::runtime::CudaStream>();
        mBufferManager = std::make_shared<tensorrt_llm::runtime::BufferManager>(mStream);
        gen.seed(42U);
    }

    static bool shouldSkip()
    {
        return false;
    }

    void setDefaultParams()
    {
        this->mTokensPerBlock = 64;
        this->mNumHeadsCompressed = 1;
        this->mNumHeadsUncompressed = 128;
        this->mLoraSize = 512;
        this->mRopeSize = 64;
        this->mUncompressedHeadSize = 128;
        this->mMaxSeqLen = 0;
        this->mMaxCachedSeqLen = 0;
        this->mMaxUncachedSeqLen = 0;
        this->mKvTokenStride = this->mNumHeadsUncompressed * this->mUncompressedHeadSize;
    }

    template <typename T>
    void fillKVOffsetData(T* arr, size_t size, bool use_both_kv = true, int max_block_per_seq = 0)
    {
        if (use_both_kv)
        {
            for (int i = 0; i < size; i++)
            {
                arr[i] = static_cast<T>(i);
            }
        }
        else
        {
            int temp_idx = 0;
            for (int i = 0; i < size; i++)
            {
                bool is_v = (((i / max_block_per_seq) % 2) == 1);
                if (is_v)
                {
                    arr[i] = static_cast<T>(0);
                }
                else
                {
                    arr[i] = static_cast<T>(temp_idx);
                    temp_idx++;
                }
            }
        }
    }

    template <typename T>
    void fillArrayDataWithMod(T* arr, size_t size)
    {
        for (int i = 0; i < size; i++)
        {
            arr[i] = static_cast<T>(i % 448);
        }
    }

    int generateRandomSizeSmallerThan(int a)
    {
        if (a <= 0)
        {
            return 0;
        }
        std::uniform_int_distribution<> distrib(0, a - 1);
        // Generate and return the random number
        return int{distrib(gen)};
    }

    template <typename T>
    void memsetZeroDevice(T* ptr, size_t size)
    {
        hipMemset(ptr, 0, size * sizeof(T));
    }

    template <typename T>
    void memsetZeroHost(T* ptr, size_t size)
    {
        std::memset(ptr, 0, size * sizeof(T));
    }

    bool allocateBuffers()
    {
        using tensorrt_llm::runtime::BufferManager;
        using tensorrt_llm::runtime::CudaStream;
        using tensorrt_llm::runtime::ITensor;
        using tensorrt_llm::runtime::bufferCast;

        auto dtype = nvinfer1::DataType::kHALF;
        if constexpr (std::is_same_v<DataType, float>)
        {
            dtype = nvinfer1::DataType::kFLOAT;
        }
        else if constexpr (std::is_same_v<DataType, half>)
        {
            dtype = nvinfer1::DataType::kHALF;
        }
        else if constexpr (std::is_same_v<DataType, __hip_bfloat16>)
        {
            dtype = nvinfer1::DataType::kBF16;
        }
        else
        {
            return false;
        }
        auto cache_dtype = dtype;
        if constexpr (std::is_same_v<TCache, __hip_fp8_e4m3_fnuz>)
        {
            cache_dtype = nvinfer1::DataType::kFP8;
            this->h_kv_scale_orig_quant
                = tensorrt_llm::runtime::BufferManager::pinned(ITensor::makeShape({1}), nvinfer1::DataType::kFLOAT);
            this->d_kv_scale_orig_quant
                = tensorrt_llm::runtime::BufferManager::gpuSync(ITensor::makeShape({1}), nvinfer1::DataType::kFLOAT);
            this->h_kv_scale_quant_orig
                = tensorrt_llm::runtime::BufferManager::pinned(ITensor::makeShape({1}), nvinfer1::DataType::kFLOAT);
            this->d_kv_scale_quant_orig
                = tensorrt_llm::runtime::BufferManager::gpuSync(ITensor::makeShape({1}), nvinfer1::DataType::kFLOAT);
            auto* kv_scale_orig_quant_ptr = bufferCast<float>(*(this->h_kv_scale_orig_quant));
            auto* kv_scale_quant_orig_ptr = bufferCast<float>(*(this->h_kv_scale_quant_orig));
            float kv_scale_orig_quant = 2.0f;
            kv_scale_orig_quant_ptr[0] = kv_scale_orig_quant;
            kv_scale_quant_orig_ptr[0] = 1.0 / kv_scale_orig_quant;
            hipMemcpy(this->d_kv_scale_orig_quant->data(), this->h_kv_scale_orig_quant->data(),
                this->h_kv_scale_orig_quant->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_kv_scale_quant_orig->data(), this->h_kv_scale_quant_orig->data(),
                this->h_kv_scale_quant_orig->getSizeInBytes(), hipMemcpyHostToDevice);
        }
        else
        {
            static_assert(std::is_same_v<DataType, TCache>, "TCache must be the same type as DataType");
        }
        this->h_cu_seq_lens = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests + 1}), nvinfer1::DataType::kINT64);
        this->h_cu_ctx_cached_kv_lens = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests + 1}), nvinfer1::DataType::kINT64);
        this->d_cu_seq_lens = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests + 1}), nvinfer1::DataType::kINT64);
        this->d_cu_ctx_cached_kv_lens = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests + 1}), nvinfer1::DataType::kINT64);
        {
            // set random sequence length
            auto* cu_seq_lens_temp_ptr = bufferCast<int64_t>(*(this->h_cu_seq_lens));
            auto* cu_ctx_cached_kv_lens_temp_ptr = bufferCast<int64_t>(*(this->h_cu_ctx_cached_kv_lens));
            cu_seq_lens_temp_ptr[0] = 0;
            cu_ctx_cached_kv_lens_temp_ptr[0] = 0;
            for (int i = 1; i <= this->mNumRequests; i++)
            {
                int temp_seq_len = generateRandomSizeSmallerThan(512);
                if (temp_seq_len <= 0)
                {
                    temp_seq_len = 1; // at least 1 token
                }
                int cached_seq_len = generateRandomSizeSmallerThan(temp_seq_len);
                this->mMaxSeqLen = std::max(temp_seq_len, this->mMaxSeqLen);
                this->mMaxCachedSeqLen = std::max(cached_seq_len, this->mMaxCachedSeqLen);
                this->mMaxUncachedSeqLen = std::max(temp_seq_len - cached_seq_len, this->mMaxUncachedSeqLen);
                this->mTotalTokens += temp_seq_len;
                this->mTotalCachedTokens += cached_seq_len;
                this->mTotalUncachedTokens += temp_seq_len - cached_seq_len;
                cu_seq_lens_temp_ptr[i] = cu_seq_lens_temp_ptr[i - 1] + temp_seq_len;
                cu_ctx_cached_kv_lens_temp_ptr[i] = cu_ctx_cached_kv_lens_temp_ptr[i - 1] + cached_seq_len;
                // std::cout << "batch " << i << "seq len: " << temp_seq_len << ", cached len: " << cached_seq_len
                //           << ", uncached len: " << temp_seq_len - cached_seq_len << std::endl;
            }
            hipMemcpy(this->d_cu_seq_lens->data(), this->h_cu_seq_lens->data(), this->h_cu_seq_lens->getSizeInBytes(),
                hipMemcpyHostToDevice);
            hipMemcpy(this->d_cu_ctx_cached_kv_lens->data(), this->h_cu_ctx_cached_kv_lens->data(),
                this->h_cu_ctx_cached_kv_lens->getSizeInBytes(), hipMemcpyHostToDevice);
        }

        // malloc kv_cache
        this->mMaxBlockPerSeq = (this->mMaxSeqLen + this->mTokensPerBlock - 1) / this->mTokensPerBlock;
        this->h_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq, this->mNumHeadsUncompressed,
                this->mTokensPerBlock, this->mUncompressedHeadSize + this->mRopeSize}),
            dtype);
        this->h_kv_cache_tensor_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq, this->mNumHeadsUncompressed,
                this->mTokensPerBlock, this->mUncompressedHeadSize + this->mRopeSize}),
            dtype);
        this->h_compressed_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 1, this->mMaxBlockPerSeq, this->mNumHeadsCompressed,
                this->mTokensPerBlock, this->mLoraSize + this->mRopeSize}),
            cache_dtype);
        this->h_compressed_kv_cache_tensor_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 1, this->mMaxBlockPerSeq, this->mNumHeadsCompressed,
                this->mTokensPerBlock, this->mLoraSize + this->mRopeSize}),
            cache_dtype);
        this->h_offset_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq}), nvinfer1::DataType::kINT32);
        this->h_compressed_offset_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq}), nvinfer1::DataType::kINT32);
        this->d_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq, this->mNumHeadsUncompressed,
                this->mTokensPerBlock, this->mUncompressedHeadSize + this->mRopeSize}),
            dtype);
        this->d_compressed_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 1, this->mMaxBlockPerSeq, this->mNumHeadsCompressed,
                this->mTokensPerBlock, this->mLoraSize + this->mRopeSize}),
            cache_dtype);
        this->d_compressed_kv_cache_tensor_ref = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 1, this->mMaxBlockPerSeq, this->mNumHeadsCompressed,
                this->mTokensPerBlock, this->mLoraSize + this->mRopeSize}),
            cache_dtype);
        this->d_offset_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq}), nvinfer1::DataType::kINT32);
        this->d_compressed_offset_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mNumRequests, 2, this->mMaxBlockPerSeq}), nvinfer1::DataType::kINT32);
        {
            auto* kv_cache_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor));
            auto* kv_cache_ref_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor_ref));
            auto* compressed_kv_cache_ptr = bufferCast<TCache>(*(this->h_compressed_kv_cache_tensor));
            auto* compressed_kv_cache_ref_ptr = bufferCast<TCache>(*(this->h_compressed_kv_cache_tensor_ref));
            auto* offset_ptr = bufferCast<int32_t>(*(this->h_offset_tensor));
            auto* compressed_offset_ptr = bufferCast<int32_t>(*(this->h_compressed_offset_tensor));
            fillArrayDataWithMod(compressed_kv_cache_ptr, this->h_compressed_kv_cache_tensor->getSize());
            fillArrayDataWithMod(compressed_kv_cache_ref_ptr, this->h_compressed_kv_cache_tensor_ref->getSize());
            memsetZeroHost<DataType>(kv_cache_ptr, this->h_kv_cache_tensor->getSize());
            memsetZeroHost<DataType>(kv_cache_ref_ptr, this->h_kv_cache_tensor_ref->getSize());
            hipMemcpy(this->d_kv_cache_tensor->data(), this->h_kv_cache_tensor->data(),
                this->h_kv_cache_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            // fillArrayDataWithMod(offset_ptr, this->offset_tensor->getSize());
            fillKVOffsetData(
                compressed_offset_ptr, this->h_compressed_offset_tensor->getSize(), false, this->mMaxBlockPerSeq);
            hipMemcpy(this->d_compressed_kv_cache_tensor->data(), this->h_compressed_kv_cache_tensor->data(),
                this->h_compressed_kv_cache_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_compressed_kv_cache_tensor_ref->data(), this->h_compressed_kv_cache_tensor_ref->data(),
                this->h_compressed_kv_cache_tensor_ref->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_compressed_offset_tensor->data(), this->h_compressed_offset_tensor->data(),
                this->h_compressed_offset_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_offset_tensor->data(), this->h_offset_tensor->data(),
                this->h_offset_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
        }

        // compressed_kv_output + k_pe_output for loadPagedKvKernel (kernel 1)
        this->h_compressed_kv_output = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, this->mLoraSize}), dtype);
        this->h_compressed_kv_output_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, this->mLoraSize}), dtype);
        this->d_compressed_kv_output = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalCachedTokens, this->mLoraSize}), dtype);
        this->h_k_pe_output = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, this->mRopeSize}), dtype);
        this->h_k_pe_output_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalCachedTokens, this->mRopeSize}), dtype);
        this->d_k_pe_output = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalCachedTokens, this->mRopeSize}), dtype);
        {
            auto* compressed_kv_output_ptr = bufferCast<DataType>(*(this->h_compressed_kv_output));
            auto* compressed_kv_output_ref_ptr = bufferCast<DataType>(*(this->h_compressed_kv_output_ref));
            memsetZeroHost<DataType>(compressed_kv_output_ptr, this->h_compressed_kv_output->getSize());
            memsetZeroHost<DataType>(compressed_kv_output_ref_ptr, this->h_compressed_kv_output_ref->getSize());
            hipMemcpy(this->d_compressed_kv_output->data(), this->h_compressed_kv_output->data(),
                this->h_compressed_kv_output->getSizeInBytes(), hipMemcpyHostToDevice);

            auto* k_pe_output_ptr = bufferCast<DataType>(*(this->h_k_pe_output));
            auto* k_pe_output_ref_ptr = bufferCast<DataType>(*(this->h_k_pe_output_ref));
            memsetZeroHost<DataType>(k_pe_output_ptr, this->h_k_pe_output->getSize());
            memsetZeroHost<DataType>(k_pe_output_ref_ptr, this->h_k_pe_output_ref->getSize());
            hipMemcpy(this->d_k_pe_output->data(), this->h_k_pe_output->data(), this->h_k_pe_output->getSizeInBytes(),
                hipMemcpyHostToDevice);
        }
        // k, v, k_pe for setPagedKvCacheForMLAKernel (kernel 2)
        this->h_k_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}), dtype);
        this->h_v_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}), dtype);
        this->h_k_pe_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsCompressed, this->mRopeSize}), dtype);
        this->d_k_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}), dtype);
        this->d_v_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsUncompressed, this->mUncompressedHeadSize}), dtype);
        this->d_k_pe_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            ITensor::makeShape({this->mTotalTokens, this->mNumHeadsCompressed, this->mRopeSize}), dtype);
        {
            auto* k_ptr = bufferCast<DataType>(*(this->h_k_tensor));
            auto* v_ptr = bufferCast<DataType>(*(this->h_v_tensor));
            auto* k_pe_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor));
            fillArrayDataWithMod(k_ptr, this->h_k_tensor->getSize());
            fillArrayDataWithMod(v_ptr, this->h_v_tensor->getSize());
            fillArrayDataWithMod(k_pe_ptr, this->h_k_pe_tensor->getSize());
            hipMemcpy(this->d_k_tensor->data(), this->h_k_tensor->data(), this->h_k_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice);
            hipMemcpy(this->d_v_tensor->data(), this->h_v_tensor->data(), this->h_v_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice);
            hipMemcpy(this->d_k_pe_tensor->data(), this->h_k_pe_tensor->data(), this->h_k_pe_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice);
        }
        return true;
    }

    void PerformLoadPagedKV()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* compressed_kv_output_ptr = bufferCast<DataType>(*(this->d_compressed_kv_output));
        auto* k_pe_output_ptr = bufferCast<DataType>(*(this->d_k_pe_output));
        auto* compressed_kv_cache_ptr = bufferCast<TCache>(*(this->d_compressed_kv_cache_tensor));
        auto* offset_ptr = bufferCast<int32_t>(*(this->d_compressed_offset_tensor));
        auto* cu_ctx_cached_kv_lens_ptr = bufferCast<int64_t>(*(this->d_cu_ctx_cached_kv_lens));
        float* kv_scale_quant_orig_ptr = nullptr;
        if constexpr (std::is_same_v<TCache, __hip_fp8_e4m3_fnuz>)
        {
            kv_scale_quant_orig_ptr = bufferCast<float>(*(this->d_kv_scale_quant_orig));
        }
        tensorrt_llm::kernels::KVBlockArray kv_cache(this->mNumRequests, this->mMaxBlockPerSeq, this->mTokensPerBlock,
            sizeof(TCache) * 1 * (this->mLoraSize + this->mRopeSize), 0, 0, 0, 0, compressed_kv_cache_ptr, nullptr,
            reinterpret_cast<tensorrt_llm::kernels::KVBlockArrayForContextFMHA::DataType*>(offset_ptr));
        tensorrt_llm::kernels::invokeMLALoadPagedKV<DataType, TCache>(compressed_kv_output_ptr, k_pe_output_ptr,
            kv_cache, this->mNumRequests, cu_ctx_cached_kv_lens_ptr, this->mMaxCachedSeqLen, this->mLoraSize,
            this->mRopeSize, kv_scale_quant_orig_ptr, this->mStream->get());
        hipStreamSynchronize(this->mStream->get());
        hipMemcpy(this->h_compressed_kv_output->data(), this->d_compressed_kv_output->data(),
            this->d_compressed_kv_output->getSizeInBytes(), hipMemcpyDeviceToHost);
        hipMemcpy(this->h_k_pe_output->data(), this->d_k_pe_output->data(), this->d_k_pe_output->getSizeInBytes(),
            hipMemcpyDeviceToHost);
    }

    void PerformLoadPagedKVRef()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* compressed_kv_output_ptr = bufferCast<DataType>(*(this->h_compressed_kv_output_ref));
        auto* k_pe_output_ptr = bufferCast<DataType>(*(this->h_k_pe_output_ref));
        auto* compressed_kv_cache_ptr = bufferCast<TCache>(*(this->h_compressed_kv_cache_tensor));
        auto* offset_ptr = bufferCast<int32_t>(*(this->h_compressed_offset_tensor));
        auto* cu_ctx_cached_kv_lens_ptr = bufferCast<int64_t>(*(this->h_cu_ctx_cached_kv_lens));
        float* kv_scale_quant_orig_ptr = nullptr;
        if constexpr (std::is_same_v<TCache, __hip_fp8_e4m3_fnuz>)
        {
            kv_scale_quant_orig_ptr = bufferCast<float>(*(this->h_kv_scale_quant_orig));
        }
        tensorrt_llm::kernels::KVBlockArray kv_cache(this->mNumRequests, this->mMaxBlockPerSeq, this->mTokensPerBlock,
            sizeof(TCache) * 1 * (this->mLoraSize + this->mRopeSize), 0, 0, 0, 0, compressed_kv_cache_ptr, nullptr,
            reinterpret_cast<tensorrt_llm::kernels::KVBlockArrayForContextFMHA::DataType*>(offset_ptr));
        loadPagedKvKernelRef<DataType, TCache>(compressed_kv_output_ptr, k_pe_output_ptr, kv_cache, this->mNumRequests,
            cu_ctx_cached_kv_lens_ptr, this->mLoraSize, this->mRopeSize, kv_scale_quant_orig_ptr);
    }

    void PerformSetPagedKV()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* k_ptr = bufferCast<DataType>(*(this->d_k_tensor));
        auto* v_ptr = bufferCast<DataType>(*(this->d_v_tensor));
        auto* k_pe_ptr = bufferCast<DataType>(*(this->d_k_pe_tensor));
        auto* kv_cache_ptr = bufferCast<DataType>(*(this->d_kv_cache_tensor));
        auto* cu_seq_lens_ptr = bufferCast<int64_t>(*(this->d_cu_seq_lens));
        tensorrt_llm::kernels::invokeMLASetPagedKV<DataType>(kv_cache_ptr, k_ptr, v_ptr, k_pe_ptr, this->mNumRequests,
            cu_seq_lens_ptr, this->mMaxSeqLen, this->mNumHeadsUncompressed, this->mUncompressedHeadSize,
            this->mRopeSize, this->mTokensPerBlock, this->mKvTokenStride, this->mStream->get());
        hipStreamSynchronize(this->mStream->get());
        hipMemcpy(this->h_kv_cache_tensor->data(), this->d_kv_cache_tensor->data(),
            this->d_kv_cache_tensor->getSizeInBytes(), hipMemcpyDeviceToHost);
    }

    void PerformSetPagedKVRef()
    {
        using tensorrt_llm::runtime::bufferCast;
        auto* k_ptr = bufferCast<DataType>(*(this->h_k_tensor));
        auto* v_ptr = bufferCast<DataType>(*(this->h_v_tensor));
        auto* k_pe_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor));
        auto* kv_cache_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor_ref));
        auto* cu_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_seq_lens));
        setPagedKvCacheForMLAKernelRef(kv_cache_ptr, k_ptr, v_ptr, k_pe_ptr, this->mNumRequests, cu_seq_lens_ptr,
            this->mMaxSeqLen, this->mNumHeadsUncompressed, this->mUncompressedHeadSize, this->mRopeSize,
            this->mTokensPerBlock, this->mKvTokenStride);
    }

    template <typename T>
    bool CheckEqual(T const* expected, T const* output, size_t size)
    {
        for (int i = 0; i < size; i++)
        {
            auto e = static_cast<float>(expected[i]);
            auto o = static_cast<float>(output[i]);
            if (!almostEqual(e, o, 1e-3, 1e-3))
            {
                TLLM_LOG_ERROR(
                    "Mismatch input value. Position of inputs: %d, expected value: %f, output value: %f", i, e, o);
                return false;
            }
        }
        return true;
    }
};

using MLATypes
    = ::testing::Types<std::pair<half, half>, std::pair<__hip_bfloat16, __hip_bfloat16>, std::pair<float, float>,
        std::pair<half, __hip_fp8_e4m3_fnuz>, std::pair<__hip_bfloat16, __hip_fp8_e4m3_fnuz>, std::pair<float, __hip_fp8_e4m3_fnuz>>;
TYPED_TEST_SUITE(MlaPreprocessTest, MLATypes);

TYPED_TEST(MlaPreprocessTest, MLAPreprocessDefault)
{
    using tensorrt_llm::runtime::bufferCast;
    using DataType = typename TestFixture::DataType;
    using TCache = typename TestFixture::TCache;
    this->mNumRequests = 8;
    this->setDefaultParams();
    EXPECT_TRUE(this->allocateBuffers());

    sync_check_cuda_error(this->mStream->get());
    bool allEqual{true};

    {
        this->PerformLoadPagedKV();
        sync_check_cuda_error(this->mStream->get());
        this->PerformLoadPagedKVRef();
        auto* compressed_kv_output_ptr = bufferCast<DataType>(*(this->h_compressed_kv_output));
        auto* k_pe_output_ptr = bufferCast<DataType>(*(this->h_k_pe_output));
        auto* compressed_kv_output_ref_ptr = bufferCast<DataType>(*(this->h_compressed_kv_output_ref));
        auto* k_pe_output_ref_ptr = bufferCast<DataType>(*(this->h_k_pe_output_ref));
        allEqual = this->CheckEqual(
            compressed_kv_output_ref_ptr, compressed_kv_output_ptr, this->h_compressed_kv_output->getSize());
        EXPECT_TRUE(allEqual);
        allEqual = this->CheckEqual(k_pe_output_ref_ptr, k_pe_output_ptr, this->h_k_pe_output->getSize());
        EXPECT_TRUE(allEqual);
    }

    {
        this->PerformSetPagedKV();
        sync_check_cuda_error(this->mStream->get());
        this->PerformSetPagedKVRef();
        auto* kv_cache_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor));
        auto* kv_cache_ref_ptr = bufferCast<DataType>(*(this->h_kv_cache_tensor_ref));
        allEqual = this->CheckEqual(kv_cache_ref_ptr, kv_cache_ptr, this->h_kv_cache_tensor->getSize());
        EXPECT_TRUE(allEqual);
    }
}
