#include "hip/hip_runtime.h"
#include <algorithm>
#include <cstdint>
#include <gtest/gtest.h>

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/quantization.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"
#include "tensorrt_llm/runtime/bufferManager.h"

#include "tensorrt_llm/kernels/mlaChunkedPrefill.cuh"
#include "tensorrt_llm/runtime/cudaStream.h"
#include <cstring>
#include <iostream>
#include <memory>
#include <random>

namespace
{
// kv_output {total_tokens, h=1, lora_size}
// k_pe_output {total_tokens, h=1, rope_size}
template <typename T>
void loadChunkedKVKernelRef(T* kv_output, T* k_pe_output, tensorrt_llm::kernels::KVBlockArray const& kv_cache,
    int num_contexts, int64_t const* cu_ctx_cached_kv_lens, int64_t const* cu_ctx_chunked_len, int const lora_size,
    int const rope_size, int const chunk_size, int const chunk_idx)
{
    int const head_size = lora_size + rope_size;
    for (int b = 0; b < num_contexts; b++)
    {
        int const kv_len = cu_ctx_cached_kv_lens[b + 1] - cu_ctx_cached_kv_lens[b];
        int const chunked_len = cu_ctx_chunked_len[b + 1] - cu_ctx_chunked_len[b];
        for (int s = 0; s < chunked_len; s++)
        {
            int const local_token_idx = chunk_idx * chunk_size + s;
            int const ld_token_offset = (cu_ctx_chunked_len[b] + s);

            auto const* kv_src = reinterpret_cast<T const*>(kv_cache.getKBlockPtr(b, local_token_idx));
            for (int d = 0; d < head_size; d++)
            {
                auto kv_block_idx = kv_cache.getKVLocalIdx(local_token_idx, 0, head_size, d);
                auto src_data = kv_src[kv_block_idx];

                if (d < lora_size)
                {
                    kv_output[ld_token_offset * lora_size + d] = src_data;
                }
                else
                {
                    k_pe_output[ld_token_offset * rope_size + (d - lora_size)] = src_data;
                }
            }
        }
    }
}

// kv {total_tokens, 2, h, nope_size}
// k_pe {total_tokens, h=1, rope_size}
// output {b, 2, ceil(max_seq / cache_tokens_per_block), h, cache_tokens_per_block, (nope_size + rope_size)}
// max_seq <= chunk_size
template <typename T>
void setChunkedKVCacheForMLAKernelRef(T* output, T* kv_ptr, T* k_pe_ptr, int num_contexts, int64_t const* cu_seq_len,
    int const max_input_seq_len, int num_heads, int nope_size, int rope_size, int cache_tokens_per_block)
{
    int head_size = nope_size + rope_size;
    int const kv_cache_size_per_block = num_heads * cache_tokens_per_block * head_size;
    int const kv_cache_block_num_per_seq = (max_input_seq_len + cache_tokens_per_block - 1) / cache_tokens_per_block;
    for (int b = 0; b < num_contexts; b++)
    {
        int const global_token_offset = cu_seq_len[b];
        int const current_seq_len = cu_seq_len[b + 1] - cu_seq_len[b];
        for (int s = 0; s < current_seq_len; s++)
        {
            int const global_token_idx = global_token_offset + s;
            int const kv_cache_block_offset_for_k
                = (b * 2 * kv_cache_block_num_per_seq + s / cache_tokens_per_block) * kv_cache_size_per_block;
            int const kv_cache_block_offset_for_v
                = kv_cache_block_offset_for_k + (kv_cache_block_num_per_seq * kv_cache_size_per_block);
            for (int h = 0; h < num_heads; h++)
            {
                int const ld_k_head_offset = (global_token_offset * 2 * num_heads * nope_size) + h * nope_size;
                int const ld_v_head_offset = ld_k_head_offset + num_heads * nope_size;
                int const ld_k_pe_head_offset = global_token_offset * rope_size;
                // copy kv
                for (int d = 0; d < nope_size; d++)
                {
                    int const ld_k_idx = ld_k_head_offset + d;
                    int const ld_v_idx = ld_v_head_offset + d;
                    int const st_k_idx = kv_cache_block_offset_for_k + h * cache_tokens_per_block * head_size
                        + s % cache_tokens_per_block * head_size + d;
                    int const st_v_idx = kv_cache_block_offset_for_v + h * cache_tokens_per_block * head_size
                        + s % cache_tokens_per_block * head_size + d;
                    output[st_k_idx] = kv_ptr[ld_k_idx];
                    output[st_v_idx] = kv_ptr[ld_v_idx];
                }

                // copy k_pe
                for (int d = 0; d < rope_size; d++)
                {
                    int const ld_k_pe_idx = ld_k_pe_head_offset + d;
                    int const st_k_pe_idx = kv_cache_block_offset_for_k + num_heads * cache_tokens_per_block * head_size
                        + h * cache_tokens_per_block * rope_size + s % cache_tokens_per_block * rope_size
                        + (rope_size + d);
                    output[st_k_pe_idx] = k_pe_ptr[ld_k_pe_idx];
                }
            }
        }
    }
}

// Q {total_q, H, D}
// KV {total_kv, 2, H, D}
// softmax_sum {total_q, H, 2} // {max/sum}
// output {total_q, H, D}
// total_q <= total_kv
template <typename T>
void selfAttentionRef(T* output, T* const Q, T* const KV, int batch_size, int num_heads, int64_t* const cu_seq_q_len,
    int64_t* const cu_seq_kv_len, int head_size, bool return_softmax, float* softmax_sum, bool causal_mask)
{
    int total_q_len = cu_seq_q_len[batch_size];
    int total_kv_len = cu_seq_kv_len[batch_size];

    for (int b = 0; b < batch_size; b++)
    {
        int curr_q_len = cu_seq_q_len[b + 1] - cu_seq_q_len[b];
        int curr_kv_len = cu_seq_kv_len[b + 1] - cu_seq_kv_len[b];
        int global_q_offset = cu_seq_q_len[b] * num_heads * head_size;
        int global_kv_offset = cu_seq_kv_len[b] * 2 * num_heads * head_size;
        int global_softmax_offset = cu_seq_q_len[b] * num_heads * 2;
        if (curr_q_len == 0 || curr_kv_len == 0)
        {
            continue; // skip empty sequences
        }
        std::vector<float> P(curr_q_len * curr_kv_len);
        for (int h = 0; h < num_heads; h++)
        {
            // BMM1
            std::fill(P.begin(), P.end(), std::numeric_limits<double>::lowest());
            T* const q_ptr = Q + global_q_offset + h * head_size;
            T* const k_ptr = KV + global_kv_offset + h * head_size;
            T* const v_ptr = k_ptr + num_heads * head_size;
            T* output_ptr = output + global_q_offset + h * head_size;
            for (int s_q = 0; s_q < curr_q_len; s_q++)
            {
                float softmax_max = std::numeric_limits<double>::lowest();
                for (int s_kv = 0; s_kv < curr_kv_len; s_kv++)
                {
                    // lower right mask
                    if (causal_mask && s_kv > curr_kv_len - curr_q_len + s_q)
                    {
                        break;
                    }
                    P[s_q * curr_kv_len + s_kv] = 0;
                    for (int d = 0; d < head_size; d++)
                    {
                        P[s_q * curr_kv_len + s_kv] += static_cast<float>(
                            q_ptr[s_q * num_heads * head_size + d] * k_ptr[s_kv * 2 * num_heads * head_size + d]);
                    }
                    if (softmax_max < P[s_q * curr_kv_len + s_kv])
                    {
                        softmax_max = P[s_q * curr_kv_len + s_kv];
                    }
                }
                for (int s_kv = 0; s_kv < curr_kv_len; s_kv++)
                {
                    // lower right mask
                    if (causal_mask && s_kv > curr_kv_len - curr_q_len + s_q)
                    {
                        break;
                    }
                    P[s_q * curr_kv_len + s_kv] -= softmax_max;
                }
                if (return_softmax)
                {
                    softmax_sum[global_softmax_offset + s_q * num_heads * 2 + h * 2] = softmax_max;
                }
            }
            // softmax
            for (int s_q = 0; s_q < curr_q_len; s_q++)
            {
                float sum = 0;
                for (int s_kv = 0; s_kv < curr_kv_len; s_kv++)
                {
                    P[s_q * curr_kv_len + s_kv] = std::exp(P[s_q * curr_kv_len + s_kv]);
                    sum += P[s_q * curr_kv_len + s_kv];
                }
                for (int s_kv = 0; s_kv < curr_kv_len; s_kv++)
                {
                    P[s_q * curr_kv_len + s_kv] /= sum;
                }
                if (return_softmax)
                {
                    softmax_sum[global_softmax_offset + s_q * num_heads * 2 + h * 2 + 1] = sum;
                }
            }
            // BMM2
            for (int s_q = 0; s_q < curr_q_len; s_q++)
            {
                for (int d = 0; d < head_size; d++)
                {
                    output_ptr[s_q * num_heads * head_size + d] = 0;
                    for (int s_kv = 0; s_kv < curr_kv_len; s_kv++)
                    {
                        output_ptr[s_q * num_heads * head_size + d] += static_cast<T>(P[s_q * curr_kv_len + s_kv]
                            * static_cast<float>(v_ptr[s_kv * 2 * num_heads * head_size + d]));
                    }
                }
            }
        }
    }
}

// chunked_KV {total_chunk_token, 2, H, D}
// KV {total_kv_token, 2, H, D}
template <typename T>
void copyRelatedChunkedKV(T* chunked_kv, T* const kv, int chunk_idx, int chunk_size, int batch_size, int num_heads,
    int64_t* const cu_kv_seq_len, int64_t* const cu_chunked_seq_len, int head_size)
{
    for (int b = 0; b < batch_size; b++)
    {
        int src_global_offset = (cu_kv_seq_len[b] + chunk_idx * chunk_size) * 2 * num_heads * head_size;
        int dst_global_offset = cu_chunked_seq_len[b] * 2 * num_heads * head_size;
        int copy_length = cu_chunked_seq_len[b + 1] - cu_chunked_seq_len[b];
        if (copy_length <= 0)
        {
            continue; // skip empty sequences
        }

        std::memcpy(chunked_kv + dst_global_offset, kv + src_global_offset,
            copy_length * 2 * num_heads * head_size * sizeof(T));
    }
}

// chunked_KV {total_chunk_token, 2, H, D}
// KV {total_kv_token, 2, H, D}
// It will copy the last chunk of KV cache to chunked_KV cache and calculate the cu_chunked_seq_len
template <typename T>
void copyFinalChunkedKV(T* chunked_kv, T* const kv, int chunk_size, int batch_size, int num_heads,
    int64_t* const cu_kv_seq_len, int64_t* cu_chunked_seq_len, int head_size, int64_t* merge_op)
{
    cu_chunked_seq_len[0] = 0;
    for (int b = 0; b < batch_size; b++)
    {
        int curr_kv_len = cu_kv_seq_len[b + 1] - cu_kv_seq_len[b];
        int last_chunk_size = curr_kv_len % chunk_size;
        if (last_chunk_size == 0)
        {
            last_chunk_size = chunk_size; // ensure at least one chunk
        }
        if (last_chunk_size == curr_kv_len)
        {
            merge_op[b] = 2; // no need to merge, just copy
        }
        else
        {
            merge_op[b] = 1;
        }
        cu_chunked_seq_len[b + 1] = cu_chunked_seq_len[b] + last_chunk_size;
        int global_token_offset = cu_kv_seq_len[b] + curr_kv_len - last_chunk_size;
        int copy_length = last_chunk_size;
        if (copy_length <= 0)
        {
            printf("copy_length is zero for batch %d, skipping...\n", b);
            continue; // skip empty sequences
        }
        int src_global_offset = global_token_offset * 2 * num_heads * head_size;
        int dst_global_offset = cu_chunked_seq_len[b] * 2 * num_heads * head_size;
        std::memcpy(chunked_kv + dst_global_offset, kv + src_global_offset,
            copy_length * 2 * num_heads * head_size * sizeof(T));
    }
}

template <typename WeightType>
float getTolerance(float scale = 1.f)
{
    float tol = 0.0;
    if constexpr (std::is_same_v<WeightType, uint8_t>)
    {
        tol = 0.1;
    }
    else if constexpr (std::is_same_v<WeightType, float>)
    {
        tol = 0.001;
    }
    else if constexpr (std::is_same_v<WeightType, half>)
    {
        tol = 0.005;
    }
    else if constexpr (std::is_same_v<WeightType, __hip_bfloat16>)
    {
        tol = 0.05;
    }
    // Keep the scale in a sane range
    return std::max(tol, scale * tol);
}
}; // namespace

template <typename _DataType>
class MlaChunkedPrefillTest : public ::testing::Test
{
protected:
    using DataType = _DataType;

    std::shared_ptr<tensorrt_llm::runtime::CudaStream> mStream;

    tensorrt_llm::runtime::BufferManager::ITensorPtr h_kv_cache_tensor{nullptr}, h_kv_cache_tensor_ref{nullptr},
        d_kv_cache_tensor{nullptr}, h_compressed_kv_cache_tensor{nullptr}, d_compressed_kv_cache_tensor{nullptr},
        h_compressed_offset_tensor{nullptr}, d_compressed_offset_tensor{nullptr}, h_cu_kv_seq_lens{nullptr},
        d_cu_kv_seq_lens{nullptr}, h_cu_chunk_lens{nullptr}, d_cu_chunk_lens{nullptr}, h_cu_q_seq_lens{nullptr},
        d_cu_q_seq_lens{nullptr},

        // for kernel 1
        h_compressed_kv_output{nullptr}, d_compressed_kv_output{nullptr}, h_k_pe_output{nullptr},
        d_k_pe_output{nullptr}, h_compressed_kv_output_ref{nullptr}, h_k_pe_output_ref{nullptr},

        // for kernel 2
        h_kv_tensor{nullptr}, d_kv_tensor{nullptr}, h_k_pe_tensor{nullptr}, d_k_pe_tensor{nullptr},

        // for merge attn {kv_full_tensor  = kv + k_pe}
        m_h_q_tensor{nullptr}, m_h_kv_full_tensor{nullptr}, m_h_chunked_kv_tensor{nullptr}, m_h_output_tensor{nullptr},
        m_h_softmax_sum_tensor{nullptr}, m_h_softmax_sum_accum_tensor{nullptr}, m_h_output_tensor_ref{nullptr},
        m_h_output_tensor_accum_ref{nullptr}, m_d_q_tensor{nullptr}, m_d_kv_full_tensor{nullptr},
        m_d_chunked_kv_tensor{nullptr}, m_d_output_tensor{nullptr}, m_d_softmax_sum_tensor{nullptr},
        m_d_softmax_sum_accum_tensor{nullptr}, m_d_output_tensor_ref{nullptr}, m_d_output_tensor_accum_ref{nullptr},
        m_h_merge_op{nullptr}, m_d_merge_op{nullptr};

    int mBatchSize{};
    int mMaxSeqLen{};
    int mMaxQSeqLen{};
    int mTotalQLen{};
    int mTotalKVLen{};
    int mChunkSize{};
    int mNumHeads{};
    int mLoraSize{};
    int mRopeSize{};
    int mNopeSize{};
    int mMaxGenLength{};
    // int mHeadSize{};
    int mTokensPerBlock{};
    int mMaxBlockPerSeq{};
    bool mIsCausalMask{};

    std::mt19937 gen;

    void SetUp() override
    {
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping mla chunked prefill test";
        }
        mStream = std::make_shared<tensorrt_llm::runtime::CudaStream>();
        gen.seed(42U);
    }

    static bool shouldSkip()
    {
        return false;
    }

    void setDefaultParams()
    {
        mBatchSize = 2;
        // mMaxSeqLen = 128;
        mChunkSize = 32;
        mNumHeads = 16;
        mLoraSize = 512;
        mRopeSize = 64;
        mNopeSize = 128;
        mIsCausalMask = false;
        mMaxGenLength = 128;
        mTokensPerBlock = 16;
    }

    void memsetZeroHost(tensorrt_llm::runtime::BufferManager::ITensorPtr& tensor)
    {
        void* ptr = tensor->data();
        std::memset(ptr, 0, tensor->getSizeInBytes());
    }

    template <typename T>
    void showHostTensor(tensorrt_llm::runtime::BufferManager::ITensorPtr& tensor)
    {
        auto* const ptr = reinterpret_cast<T*>(tensor->data());
        for (int _ = 0; _ < tensor->getSize(); _++)
        {
            std::cout << static_cast<float>(ptr[_]) << " ";
        }
        std::cout << std::endl;
    }

    int generateRandomSizeSmallerThan(int a)
    {
        if (a <= 0)
        {
            return 0;
        }
        std::uniform_int_distribution<> distrib(0, a - 1);
        // Generate and return the random number
        return int{distrib(gen)};
    }

    float generateRandomFloat(float min, float max)
    {
        std::uniform_real_distribution<float> dist(min, max);
        return dist(gen);
    }

    template <typename T>
    void generateRandomData(T* data, int size)
    {
        for (int i = 0; i < size; i++)
        {
            data[i] = static_cast<T>(generateRandomFloat(-1.0f, 1.0f));
        }
    }

    template <typename T>
    void fillKVOffsetData(T* arr, size_t size, bool use_both_kv = true, int max_block_per_seq = 0)
    {
        if (use_both_kv)
        {
            for (int i = 0; i < size; i++)
            {
                arr[i] = static_cast<T>(i);
            }
        }
        else
        {
            int temp_idx = 0;
            for (int i = 0; i < size; i++)
            {
                bool is_v = (((i / max_block_per_seq) % 2) == 1);
                if (is_v)
                {
                    arr[i] = static_cast<T>(0);
                }
                else
                {
                    arr[i] = static_cast<T>(temp_idx);
                    temp_idx++;
                }
            }
        }
    }

    template <typename T>
    void fillArrayDataWithMod(T* arr, size_t size)
    {
        for (int i = 0; i < size; i++)
        {
            arr[i] = static_cast<T>(i % 448);
        }
    }

    bool allocateBuffers()
    {
        using tensorrt_llm::runtime::BufferManager;
        using tensorrt_llm::runtime::CudaStream;
        using tensorrt_llm::runtime::ITensor;
        using tensorrt_llm::runtime::bufferCast;

        auto dtype = nvinfer1::DataType::kHALF;
        if constexpr (std::is_same_v<DataType, float>)
        {
            dtype = nvinfer1::DataType::kFLOAT;
        }
        else if constexpr (std::is_same_v<DataType, half>)
        {
            dtype = nvinfer1::DataType::kHALF;
        }
        else if constexpr (std::is_same_v<DataType, __hip_bfloat16>)
        {
            dtype = nvinfer1::DataType::kBF16;
        }
        else
        {
            return false;
        }

        // cu lens
        this->h_cu_kv_seq_lens = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize + 1}), nvinfer1::DataType::kINT64);
        this->h_cu_chunk_lens = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize + 1}), nvinfer1::DataType::kINT64);
        this->h_cu_q_seq_lens = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize + 1}), nvinfer1::DataType::kINT64);
        this->d_cu_kv_seq_lens = tensorrt_llm::runtime::BufferManager::gpuSync(
            this->h_cu_kv_seq_lens->getShape(), nvinfer1::DataType::kINT64);
        this->d_cu_chunk_lens = tensorrt_llm::runtime::BufferManager::gpuSync(
            this->h_cu_chunk_lens->getShape(), nvinfer1::DataType::kINT64);
        this->d_cu_q_seq_lens = tensorrt_llm::runtime::BufferManager::gpuSync(
            this->h_cu_q_seq_lens->getShape(), nvinfer1::DataType::kINT64);
        {
            this->mMaxSeqLen = 0;
            this->mMaxQSeqLen = 0;
            this->mTotalQLen = 0;
            this->mTotalKVLen = 0;
            // we only initialize cu_seq_lens
            auto* cu_kv_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_kv_seq_lens));
            auto* cu_q_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_q_seq_lens));
            cu_kv_seq_lens_ptr[0] = 0;
            cu_q_seq_lens_ptr[0] = 0;
            for (int i = 0; i < this->mBatchSize; i++)
            {
                int temp_seq_len = this->generateRandomSizeSmallerThan(this->mMaxGenLength);
                if (temp_seq_len == 0)
                {
                    temp_seq_len = 1; // ensure at least one token
                }
                this->mMaxSeqLen = std::max(this->mMaxSeqLen, temp_seq_len);
                cu_kv_seq_lens_ptr[i + 1] = cu_kv_seq_lens_ptr[i] + temp_seq_len;
                auto temp_q_seq_len = temp_seq_len % this->mChunkSize;
                if (temp_q_seq_len == 0)
                {
                    temp_q_seq_len = this->mChunkSize; // ensure at least one chunk
                }
                cu_q_seq_lens_ptr[i + 1] = cu_q_seq_lens_ptr[i] + temp_q_seq_len;
                this->mMaxQSeqLen = std::max(this->mMaxQSeqLen, temp_q_seq_len);
                this->mTotalQLen += temp_q_seq_len;
                this->mTotalKVLen += temp_seq_len;
            }
            hipMemcpy(this->d_cu_kv_seq_lens->data(), this->h_cu_kv_seq_lens->data(),
                this->h_cu_kv_seq_lens->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_cu_q_seq_lens->data(), this->h_cu_q_seq_lens->data(),
                this->h_cu_q_seq_lens->getSizeInBytes(), hipMemcpyHostToDevice);
        }
        // kv cache
        this->mMaxBlockPerSeq = (this->mMaxSeqLen + this->mTokensPerBlock - 1) / this->mTokensPerBlock;
        int maxChunkBlockPerSeq = (this->mChunkSize + this->mTokensPerBlock - 1) / this->mTokensPerBlock;
        this->h_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize, 2, maxChunkBlockPerSeq, this->mNumHeads, this->mTokensPerBlock,
                this->mNopeSize + this->mRopeSize}),
            dtype);

        this->h_kv_cache_tensor_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize, 2, maxChunkBlockPerSeq, this->mNumHeads, this->mTokensPerBlock,
                this->mNopeSize + this->mRopeSize}),
            dtype);

        this->h_compressed_kv_cache_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize, 2, this->mMaxBlockPerSeq, this->mNumHeads, this->mTokensPerBlock,
                this->mLoraSize + this->mRopeSize}),
            dtype);
        this->h_compressed_offset_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize, 2, this->mMaxBlockPerSeq + 1}), nvinfer1::DataType::kINT32);
        this->d_kv_cache_tensor
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->h_kv_cache_tensor->getShape(), dtype);
        this->d_compressed_kv_cache_tensor
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->h_compressed_kv_cache_tensor->getShape(), dtype);
        this->d_compressed_offset_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            this->h_compressed_offset_tensor->getShape(), nvinfer1::DataType::kINT32);

        {
            auto* compressed_kv_cache_ptr = bufferCast<DataType>(*(this->h_compressed_kv_cache_tensor));
            auto* offset_ptr = bufferCast<int32_t>(*(this->h_compressed_offset_tensor));

            this->memsetZeroHost(this->h_kv_cache_tensor);
            this->memsetZeroHost(this->h_kv_cache_tensor_ref);

            this->fillArrayDataWithMod(compressed_kv_cache_ptr, this->h_compressed_kv_cache_tensor->getSize());
            this->fillKVOffsetData(
                offset_ptr, this->h_compressed_offset_tensor->getSize(), false, this->mMaxBlockPerSeq);
            hipMemcpy(this->d_kv_cache_tensor->data(), this->h_kv_cache_tensor->data(),
                this->h_kv_cache_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_compressed_kv_cache_tensor->data(), this->h_compressed_kv_cache_tensor->data(),
                this->h_compressed_kv_cache_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_compressed_offset_tensor->data(), this->h_compressed_offset_tensor->data(),
                this->h_compressed_offset_tensor->getSizeInBytes(), hipMemcpyHostToDevice);
        }

        // tensor
        // kv, k_pe for invokeMLALoadChunkedKV (kernel 1)
        this->h_compressed_kv_output = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize * this->mChunkSize, 1, this->mLoraSize}), dtype);
        this->h_k_pe_output = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize * this->mChunkSize, 1, this->mRopeSize}), dtype);
        this->h_compressed_kv_output_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize * this->mChunkSize, 1, this->mLoraSize}), dtype);
        this->h_k_pe_output_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize * this->mChunkSize, 1, this->mRopeSize}), dtype);
        this->d_compressed_kv_output
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->h_compressed_kv_output->getShape(), dtype);
        this->d_k_pe_output = tensorrt_llm::runtime::BufferManager::gpuSync(this->h_k_pe_output->getShape(), dtype);
        {
            this->memsetZeroHost(this->h_compressed_kv_output);
            this->memsetZeroHost(this->h_k_pe_output);
            this->memsetZeroHost(this->h_compressed_kv_output_ref);
            this->memsetZeroHost(this->h_k_pe_output_ref);

            hipMemcpy(this->d_compressed_kv_output->data(), this->h_compressed_kv_output->data(),
                this->h_compressed_kv_output->getSizeInBytes(), hipMemcpyHostToDevice);
            hipMemcpy(this->d_k_pe_output->data(), this->h_k_pe_output->data(), this->h_k_pe_output->getSizeInBytes(),
                hipMemcpyHostToDevice);
        }

        // kv, k_pe for invokeMLASetChunkedKV (kernel 2)
        this->h_kv_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize * this->mChunkSize, 2, this->mNumHeads, this->mNopeSize}), dtype);
        this->h_k_pe_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize * this->mChunkSize, 1, this->mRopeSize}), dtype);
        this->d_kv_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(this->h_kv_tensor->getShape(), dtype);
        this->d_k_pe_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(this->h_k_pe_tensor->getShape(), dtype);
        {
            auto* kv_ptr = bufferCast<DataType>(*(this->h_kv_tensor));
            auto* k_pe_ptr = bufferCast<DataType>(*(this->h_k_pe_tensor));

            generateRandomData(kv_ptr, h_kv_tensor->getSize());
            generateRandomData(k_pe_ptr, h_k_pe_tensor->getSize());

            hipMemcpyAsync(d_kv_tensor->data(), h_kv_tensor->data(), h_kv_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(d_k_pe_tensor->data(), h_k_pe_tensor->data(), h_k_pe_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice, mStream->get());
        }

        // invokeMergeAttnWithSoftmax, we just ignore rope_size here for simplicity

        this->m_h_q_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalQLen, this->mNumHeads, this->mNopeSize}), dtype);
        this->m_h_kv_full_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalKVLen, 2, this->mNumHeads, this->mNopeSize}), dtype);
        this->m_h_chunked_kv_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize * this->mChunkSize, 2, this->mNumHeads, this->mNopeSize}), dtype);
        this->m_h_output_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalQLen, this->mNumHeads, this->mNopeSize}), dtype);
        this->m_h_softmax_sum_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({2, this->mTotalQLen, this->mNumHeads}), nvinfer1::DataType::kFLOAT);
        this->m_h_softmax_sum_accum_tensor = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({2, this->mTotalQLen, this->mNumHeads}), nvinfer1::DataType::kFLOAT);
        this->m_h_output_tensor_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalQLen, this->mNumHeads, this->mNopeSize}), dtype);
        this->m_h_output_tensor_accum_ref = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mTotalQLen, this->mNumHeads, this->mNopeSize}), dtype);
        this->m_h_merge_op = tensorrt_llm::runtime::BufferManager::pinned(
            ITensor::makeShape({this->mBatchSize}), nvinfer1::DataType::kINT64);
        this->m_d_q_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(this->m_h_q_tensor->getShape(), dtype);
        this->m_d_kv_full_tensor
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->m_h_kv_full_tensor->getShape(), dtype);
        this->m_d_chunked_kv_tensor
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->m_h_chunked_kv_tensor->getShape(), dtype);
        this->m_d_output_tensor
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->m_h_output_tensor->getShape(), dtype);
        this->m_d_softmax_sum_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            this->m_h_softmax_sum_tensor->getShape(), nvinfer1::DataType::kFLOAT);
        this->m_d_softmax_sum_accum_tensor = tensorrt_llm::runtime::BufferManager::gpuSync(
            this->m_h_softmax_sum_accum_tensor->getShape(), nvinfer1::DataType::kFLOAT);
        this->m_d_output_tensor_ref
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->m_h_output_tensor_ref->getShape(), dtype);
        this->m_d_output_tensor_accum_ref
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->m_h_output_tensor_accum_ref->getShape(), dtype);
        this->m_d_merge_op
            = tensorrt_llm::runtime::BufferManager::gpuSync(this->m_h_merge_op->getShape(), nvinfer1::DataType::kINT64);

        {
            auto* q_ptr = bufferCast<DataType>(*(this->m_h_q_tensor));
            auto* kv_ptr = bufferCast<DataType>(*(this->m_h_kv_full_tensor));

            generateRandomData(q_ptr, m_h_q_tensor->getSize());
            generateRandomData(kv_ptr, m_h_kv_full_tensor->getSize());
            this->memsetZeroHost(m_h_chunked_kv_tensor);
            this->memsetZeroHost(m_h_output_tensor);
            this->memsetZeroHost(m_h_softmax_sum_tensor);
            this->memsetZeroHost(m_h_softmax_sum_accum_tensor);
            this->memsetZeroHost(m_h_output_tensor_ref);
            this->memsetZeroHost(m_h_output_tensor_accum_ref);

            // Copy data to device
            hipMemcpyAsync(m_d_q_tensor->data(), m_h_q_tensor->data(), m_h_q_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(m_d_kv_full_tensor->data(), m_h_kv_full_tensor->data(),
                m_h_kv_full_tensor->getSizeInBytes(), hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(m_d_chunked_kv_tensor->data(), m_h_chunked_kv_tensor->data(),
                m_h_chunked_kv_tensor->getSizeInBytes(), hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(m_d_output_tensor->data(), m_h_output_tensor->data(), m_h_output_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(m_d_softmax_sum_tensor->data(), m_h_softmax_sum_tensor->data(),
                m_h_softmax_sum_tensor->getSizeInBytes(), hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(m_d_softmax_sum_accum_tensor->data(), m_h_softmax_sum_accum_tensor->data(),
                m_h_softmax_sum_accum_tensor->getSizeInBytes(), hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(m_d_output_tensor_ref->data(), m_h_output_tensor_ref->data(),
                m_h_output_tensor_ref->getSizeInBytes(), hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(m_d_output_tensor_accum_ref->data(), m_h_output_tensor_accum_ref->data(),
                m_h_output_tensor_accum_ref->getSizeInBytes(), hipMemcpyHostToDevice, mStream->get());
            hipStreamSynchronize(mStream->get());
        }
        return true;
    }

    void PerformNormalAttention()
    {
        using tensorrt_llm::runtime::bufferCast;

        auto* q_ptr = bufferCast<DataType>(*(this->m_h_q_tensor));
        auto* kv_ptr = bufferCast<DataType>(*(this->m_h_kv_full_tensor));
        auto* output_ptr = bufferCast<DataType>(*(this->m_h_output_tensor));
        auto* cu_q_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_q_seq_lens));
        auto* cu_kv_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_kv_seq_lens));
        selfAttentionRef(output_ptr, q_ptr, kv_ptr, this->mBatchSize, this->mNumHeads, cu_q_seq_lens_ptr,
            cu_kv_seq_lens_ptr, this->mNopeSize, false, nullptr, this->mIsCausalMask);
    }

    void PerformMergedAttention()
    {
        using tensorrt_llm::runtime::bufferCast;

        auto* h_q_ptr = bufferCast<DataType>(*(this->m_h_q_tensor));
        auto* h_kv_ptr = bufferCast<DataType>(*(this->m_h_kv_full_tensor));
        auto* h_chunked_kv_ptr = bufferCast<DataType>(*(this->m_h_chunked_kv_tensor));
        auto* h_output_ptr = bufferCast<DataType>(*(this->m_h_output_tensor_ref));
        auto* h_output_accum_ptr = bufferCast<DataType>(*(this->m_h_output_tensor_accum_ref));
        auto* h_softmax_sum_ptr = bufferCast<float>(*(this->m_h_softmax_sum_tensor));
        auto* h_softmax_sum_accum_ptr = bufferCast<float>(*(this->m_h_softmax_sum_accum_tensor));
        auto* h_cu_q_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_q_seq_lens));
        auto* h_cu_kv_seq_lens_ptr = bufferCast<int64_t>(*(this->h_cu_kv_seq_lens));
        auto* h_cu_chunk_lens_ptr = bufferCast<int64_t>(*(this->h_cu_chunk_lens));
        auto* h_merge_op = bufferCast<int64_t>(*(this->m_h_merge_op));
        auto* d_kv_ptr = bufferCast<DataType>(*(this->m_d_kv_full_tensor));
        auto* d_chunked_kv_ptr = bufferCast<DataType>(*(this->m_d_chunked_kv_tensor));
        auto* d_softmax_sum_ptr = bufferCast<float>(*(this->m_d_softmax_sum_tensor));
        auto* d_softmax_sum_accum_ptr = bufferCast<float>(*(this->m_d_softmax_sum_accum_tensor));
        auto* d_output_ptr = bufferCast<DataType>(*(this->m_d_output_tensor_ref));
        auto* d_output_accum_ptr = bufferCast<DataType>(*(this->m_d_output_tensor_accum_ref));
        auto* d_merge_op = bufferCast<int64_t>(*(this->m_d_merge_op));
        auto* d_cu_q_seq_lens_ptr = bufferCast<int64_t>(*(this->d_cu_q_seq_lens));

        int const loop_count = (this->mMaxSeqLen + this->mChunkSize - 1) / this->mChunkSize;
        // do not apply mask
        for (int _ = 0; _ < loop_count - 1; _++)
        {
            // get chunked len for each request
            h_cu_chunk_lens_ptr[0] = 0;
            for (int b = 0; b < this->mBatchSize; b++)
            {
                int curr_kv_len = h_cu_kv_seq_lens_ptr[b + 1] - h_cu_kv_seq_lens_ptr[b];
                int used_kv_len = loop_count * this->mChunkSize;
                int curr_chunk_len = std::min(this->mChunkSize, curr_kv_len - used_kv_len);
                if (curr_chunk_len != this->mChunkSize)
                {
                    // last chunk, we should skip it.
                    curr_chunk_len = 0;
                }
                else
                {
                    if (used_kv_len + curr_chunk_len == curr_kv_len)
                    {
                        // last chunk, we should skip it.
                        curr_chunk_len = 0;
                    }
                }
                h_cu_chunk_lens_ptr[b + 1] = h_cu_chunk_lens_ptr[b] + curr_chunk_len;
                if (_ == 0 && curr_chunk_len > 0)
                {
                    h_merge_op[b] = 2; // only copy result
                }
                else if (curr_chunk_len > 0)
                {
                    h_merge_op[b] = 1; // merge result
                }
                else
                {
                    h_merge_op[b] = 0; // skip
                }
            }
            hipMemcpy(d_merge_op, h_merge_op, this->m_h_merge_op->getSizeInBytes(), hipMemcpyHostToDevice);
            // copy related kv chunk data
            copyRelatedChunkedKV(h_chunked_kv_ptr, h_kv_ptr, _, this->mChunkSize, this->mBatchSize, this->mNumHeads,
                h_cu_kv_seq_lens_ptr, h_cu_chunk_lens_ptr, this->mNopeSize);
            // attention
            selfAttentionRef<DataType>(h_output_ptr, h_q_ptr, h_chunked_kv_ptr, this->mBatchSize, this->mNumHeads,
                h_cu_q_seq_lens_ptr, h_cu_chunk_lens_ptr, this->mNopeSize, true, h_softmax_sum_ptr, false);
            // merge attention

            // copy curr_attn and softmax_sum to device
            hipMemcpyAsync(d_softmax_sum_accum_ptr, h_softmax_sum_accum_ptr,
                this->m_h_softmax_sum_accum_tensor->getSizeInBytes(), hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(d_softmax_sum_ptr, h_softmax_sum_ptr, this->m_h_softmax_sum_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(d_output_accum_ptr, h_output_accum_ptr, this->m_h_output_tensor_accum_ref->getSizeInBytes(),
                hipMemcpyHostToDevice, mStream->get());
            hipMemcpyAsync(d_output_ptr, h_output_ptr, this->m_h_output_tensor->getSizeInBytes(),
                hipMemcpyHostToDevice, mStream->get());
            sync_check_cuda_error(mStream->get());
            // merge softmax
            tensorrt_llm::kernels::invokeMergeAttnWithSoftmax<DataType>(d_output_accum_ptr, d_softmax_sum_accum_ptr,
                d_output_accum_ptr, d_softmax_sum_accum_ptr, d_output_ptr, d_softmax_sum_ptr, this->mBatchSize,
                d_cu_q_seq_lens_ptr, this->mMaxQSeqLen, d_merge_op, this->mNumHeads, this->mNopeSize, mStream->get());
            sync_check_cuda_error(mStream->get());
            // copy merged softmax sum back to host
            hipMemcpyAsync(h_softmax_sum_accum_ptr, d_softmax_sum_accum_ptr,
                this->m_h_softmax_sum_tensor->getSizeInBytes(), hipMemcpyDeviceToHost, mStream->get());
            hipMemcpyAsync(h_output_accum_ptr, d_output_accum_ptr, this->m_h_output_tensor->getSizeInBytes(),
                hipMemcpyDeviceToHost, mStream->get());
            sync_check_cuda_error(mStream->get());
        }
        // final round, apply causal mask.

        // copy the last chunked kv data
        copyFinalChunkedKV<DataType>(h_chunked_kv_ptr, h_kv_ptr, this->mChunkSize, this->mBatchSize, this->mNumHeads,
            h_cu_kv_seq_lens_ptr, h_cu_chunk_lens_ptr, this->mNopeSize, h_merge_op);
        // attention
        selfAttentionRef<DataType>(h_output_ptr, h_q_ptr, h_chunked_kv_ptr, this->mBatchSize, this->mNumHeads,
            h_cu_q_seq_lens_ptr, h_cu_chunk_lens_ptr, this->mNopeSize, true, h_softmax_sum_ptr, this->mIsCausalMask);
        // merge attention
        // copy curr_attn and softmax_sum to device
        hipMemcpyAsync(d_softmax_sum_accum_ptr, h_softmax_sum_accum_ptr,
            this->m_h_softmax_sum_accum_tensor->getSizeInBytes(), hipMemcpyHostToDevice, mStream->get());
        hipMemcpyAsync(d_softmax_sum_ptr, h_softmax_sum_ptr, this->m_h_softmax_sum_tensor->getSizeInBytes(),
            hipMemcpyHostToDevice, mStream->get());
        hipMemcpyAsync(d_output_accum_ptr, h_output_accum_ptr, this->m_h_output_tensor_accum_ref->getSizeInBytes(),
            hipMemcpyHostToDevice, mStream->get());
        hipMemcpyAsync(d_output_ptr, h_output_ptr, this->m_h_output_tensor->getSizeInBytes(), hipMemcpyHostToDevice,
            mStream->get());
        sync_check_cuda_error(mStream->get());
        tensorrt_llm::kernels::invokeMergeAttnWithSoftmax<DataType>(d_output_accum_ptr, d_softmax_sum_accum_ptr,
            d_output_accum_ptr, d_softmax_sum_accum_ptr, d_output_ptr, d_softmax_sum_ptr, this->mBatchSize,
            d_cu_q_seq_lens_ptr, this->mMaxQSeqLen, d_merge_op, this->mNumHeads, this->mNopeSize, mStream->get());
        sync_check_cuda_error(mStream->get());
        // copy merged softmax sum back to host
        hipMemcpyAsync(h_softmax_sum_accum_ptr, d_softmax_sum_accum_ptr,
            this->m_h_softmax_sum_tensor->getSizeInBytes(), hipMemcpyDeviceToHost, mStream->get());
        hipMemcpyAsync(h_output_accum_ptr, d_output_accum_ptr, this->m_h_output_tensor->getSizeInBytes(),
            hipMemcpyDeviceToHost, mStream->get());
        sync_check_cuda_error(mStream->get());
    }
};

using MLATypes = ::testing::Types<half, __hip_bfloat16, float>;
TYPED_TEST_SUITE(MlaChunkedPrefillTest, MLATypes);

TYPED_TEST(MlaChunkedPrefillTest, MlaChunkedPrefillDefault)
{
    using tensorrt_llm::runtime::bufferCast;
    using DataType = typename TestFixture::DataType;
    this->setDefaultParams();
    this->allocateBuffers();

    sync_check_cuda_error(this->mStream->get());
    bool allEqual{true};

    this->PerformNormalAttention();
    sync_check_cuda_error(this->mStream->get());

    this->PerformMergedAttention();
    sync_check_cuda_error(this->mStream->get());

    // check result
    auto* output_ptr = bufferCast<DataType>(*(this->m_h_output_tensor));
    auto* output_ref_ptr = bufferCast<DataType>(*(this->m_h_output_tensor_accum_ref));
    for (int i = 0; i < this->m_h_output_tensor->getSize(); i++)
    {
        if (std::abs(static_cast<float>(output_ptr[i]) - static_cast<float>(output_ref_ptr[i]))
            > getTolerance<DataType>(output_ptr[i]))
        {
            std::cout << "Output mismatch at index " << i << ": "
                      << "expected " << static_cast<float>(output_ref_ptr[i]) << ", got "
                      << static_cast<float>(output_ptr[i]) << std::endl;
            allEqual = false;
            break;
        }
    }
    ASSERT_TRUE(allEqual);
}

TYPED_TEST(MlaChunkedPrefillTest, MlaChunkedPrefillCausalMask)
{
    using tensorrt_llm::runtime::bufferCast;
    using DataType = typename TestFixture::DataType;
    this->setDefaultParams();
    this->mIsCausalMask = true;
    this->allocateBuffers();

    sync_check_cuda_error(this->mStream->get());
    bool allEqual{true};

    this->PerformNormalAttention();
    sync_check_cuda_error(this->mStream->get());

    this->PerformMergedAttention();
    sync_check_cuda_error(this->mStream->get());

    // check result
    auto* output_ptr = bufferCast<DataType>(*(this->m_h_output_tensor));
    auto* output_ref_ptr = bufferCast<DataType>(*(this->m_h_output_tensor_accum_ref));
    for (int i = 0; i < this->h_output_tensor->getSize(); i++)
    {
        if (std::abs(static_cast<float>(output_ptr[i]) - static_cast<float>(output_ref_ptr[i]))
            > getTolerance<DataType>(output_ptr[i]))
        {
            std::cout << "Output mismatch at index " << i << ": "
                      << "expected " << static_cast<float>(output_ref_ptr[i]) << ", got "
                      << static_cast<float>(output_ptr[i]) << std::endl;
            allEqual = false;
            break;
        }
    }
    ASSERT_TRUE(allEqual);
}
