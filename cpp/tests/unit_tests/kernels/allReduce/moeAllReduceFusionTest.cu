#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <nccl.h>

#include <cstdint>
#include <functional>
#include <iostream>
#include <random>
#include <vector>

#include "tensorrt_llm/kernels/communicationKernels/allReduceWorkspace.h"
#include "tensorrt_llm/kernels/communicationKernels/moeAllReduceFusionKernels.h"
#include "tensorrt_llm/kernels/quantization.h"
#include "tensorrt_llm/kernels/rmsnormKernels.h"
#include "tensorrt_llm/runtime/cudaStream.h"
#include "tensorrt_llm/runtime/utils/mpiUtils.h"
#include "tensorrt_llm/runtime/utils/multiDeviceUtils.h"

namespace mpi = tensorrt_llm::mpi;
namespace tr = tensorrt_llm::runtime;
using namespace tensorrt_llm::kernels;

template <typename DType>
__global__ void residual_add_kernel(DType* data, DType* residual, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;
    data[idx] = data[idx] + residual[idx];
}

template <typename DType>
void residual_add(DType* data, DType* residual, int size, hipStream_t stream)
{
    residual_add_kernel<<<size / 128, 128, 0, stream>>>(data, residual, size);
}

template <typename DType>
__global__ void cast_to_fp32_kernel(DType* in, float* out, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;
    out[idx] = static_cast<float>(in[idx]);
}

template <typename DType>
void cast_to_fp32(DType* in, float* out, int size, hipStream_t stream)
{
    cast_to_fp32_kernel<<<size / 128, 128, 0, stream>>>(in, out, size);
}

template <typename T>
void print(int rank, void* _pa, int size)
{
    auto pa = reinterpret_cast<T*>(_pa);
    if (rank == 0)
    {
        printf("print: [");
        for (int n = 0; n < 20; ++n)
        {
            float v = static_cast<float>(pa[n]);
            printf("%f, ", v);
        }
        printf("...]\n");
    }
}

template <typename T>
float compare(int rank, void* _pa, void* _pb, int size, float scale, std::string const& cmp_info = "")
{
    auto pa = reinterpret_cast<T*>(_pa);
    auto pb = reinterpret_cast<T*>(_pb);
    float max_diff = 0.f, tot_diff = 0.f;
    float max_val = 0.f;
    int diff_cnt = 0;
    float threshold = 1e-7;
    static char* ar_debug = std::getenv("AR_DEBUG");
    if (ar_debug && rank == 0)
    {
        printf("TensorA: [");
        for (int n = 0; n < 20; ++n)
        {
            float v = static_cast<float>(pa[n]);
            printf("%f, ", v);
        }
        printf("...]\n");
        printf("TensorB: [");
        for (int n = 0; n < 20; ++n)
        {
            float v = static_cast<float>(pb[n]);
            printf("%f, ", v);
        }
        printf("...]\n");
    }
    int print_cnt = 0;
    for (int n = 0; n < size; ++n)
    {
        float va = static_cast<float>(pa[n]);
        float vb = static_cast<float>(pb[n]);
        max_val = std::max(max_val, vb);
        float diff = std::abs(va - vb);
        if (diff > threshold)
        {
            max_diff = std::max(max_diff, diff);
            tot_diff += diff;
            ++diff_cnt;
        }
        if (rank == 0 && print_cnt < 20 && ar_debug && diff / (std::abs(vb) + 1e-7) > 0.1)
        {
            ++print_cnt;
            printf("idx %d, va %f, vb %f\n", n, va, vb);
        }
    }
    float diff_thres = max_val * scale;
    if (rank == 0)
    {
        TLLM_LOG_INFO("[%s] rank %d, max diff %f (diff threshold %f), avg diff %f, diff cnt %d/%d", cmp_info.c_str(),
            rank, max_diff, diff_thres, tot_diff / std::max(diff_cnt, 1), diff_cnt, size);
    }
    return max_diff <= diff_thres;
}

template <typename T1, typename T2>
void random_fill(T1* data, int size, T2 minv, T2 maxv)
{
    static int rseed = 20250227;
    std::mt19937 gen(rseed++);
    std::uniform_real_distribution<float> dis(static_cast<float>(minv), static_cast<float>(maxv));
    for (int i = 0; i < size; ++i)
    {
        data[i] = static_cast<T1>(dis(gen));
    }
}

struct CudaBuffer
{
    void* m_d_data;
    void* m_h_data;
    int m_size;

    CudaBuffer(int size_in_bytes = 0)
        : m_size(size_in_bytes)
        , m_d_data(nullptr)
        , m_h_data(nullptr)
    {
        allocate(size_in_bytes);
    }

    void allocate(int size_in_bytes)
    {
        if (size_in_bytes == 0)
            return;
        TLLM_CHECK(m_d_data == nullptr && m_h_data == nullptr);
        m_size = size_in_bytes;
        TLLM_CUDA_CHECK(hipMalloc(&m_d_data, m_size));
        TLLM_CUDA_CHECK(hipMemset(m_d_data, 0, m_size));
        m_h_data = malloc(m_size);
    }

    template <typename T = void>
    T* device_data()
    {
        TLLM_CHECK(m_d_data != nullptr);
        return reinterpret_cast<T*>(m_d_data);
    }

    template <typename T = void>
    T* host_data()
    {
        TLLM_CHECK(m_h_data != nullptr);
        d2h();
        return reinterpret_cast<T*>(m_h_data);
    }

    template <typename DType, typename VType>
    void random(VType minv, VType maxv)
    {
        random_fill(reinterpret_cast<DType*>(m_h_data), m_size / sizeof(DType), minv, maxv);
        h2d();
    }

    void h2d()
    {
        TLLM_CUDA_CHECK(hipMemcpy(m_d_data, m_h_data, m_size, hipMemcpyHostToDevice));
    }

    void d2h()
    {
        TLLM_CUDA_CHECK(hipMemcpy(m_h_data, m_d_data, m_size, hipMemcpyDeviceToHost));
    }

    ~CudaBuffer()
    {
        if (m_d_data)
        {
            TLLM_CUDA_CHECK(hipFree(m_d_data));
        }
        if (m_h_data)
        {
            free(m_h_data);
        }
    }
};

/////////////////////////////////////////////////////////////////
//                  * MoE Reduction Fusion *                   //
/////////////////////////////////////////////////////////////////

template <typename IOType>
union ACCESS_TYPE
{
    static constexpr int ELEM_PER_ACCESS = 16 / sizeof(IOType);

    // For LDG.128 STG.128 access
    int4 packed;
    IOType unpacked[ELEM_PER_ACCESS];
};

template <typename IOType, typename ScaleType>
__global__ void moe_reduction_kernel(IOType const* ggemm2_actexp_m_hidden_in, IOType const* fc2_m_hidden_in,
    ScaleType const* scale_actexp_m_in, int const* actexpi_to_global_expid, IOType* reduce_m_hidden_ou, int num_act_exp,
    int num_token, int hidden_size)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))

    static_assert(sizeof(ScaleType) >= sizeof(IOType), "This kernel assume scale type is more precious than io type");
    namespace cg = cooperative_groups;
    cg::cluster_group cluster = cg::this_cluster();
    cg::grid_group grid = cg::this_grid();

    using ACC_TYPE = ACCESS_TYPE<IOType>;

    // Each cluster handle one token
    // Each thread handle ACC_TYPE::ELEM_PER_ACCESS element per token per expert

    int threadid_in_cluster = cluster.thread_rank();
    // Start Offset within one token's hidden_size of element
    // Current thread handle token[thread_offset_within_token : thread_offset_within_token + ACC_TYPE::ELEM_PER_ACCESS]
    int thread_offset_within_token = threadid_in_cluster * ACC_TYPE::ELEM_PER_ACCESS;

    if (thread_offset_within_token >= hidden_size)
    {
        return;
    }

    cudaGridDependencySynchronize();

    // Same as AR + Fusion kernel, use persistent kernel design
    for (int token_id = grid.cluster_rank(); token_id < num_token; token_id += grid.num_clusters())
    {

        // Offset within (num_token, hidden_size) in unit of element
        int thread_offset_across_token = token_id * hidden_size + thread_offset_within_token;

        ACC_TYPE accumulator;
#pragma unroll
        for (int i = 0; i < ACC_TYPE::ELEM_PER_ACCESS; ++i)
        {
            accumulator.unpacked[i] = static_cast<IOType>(0);
        }

        // * Iterate through all active expert
        for (int actexp_i = 0; actexp_i < num_act_exp; ++actexp_i)
        {

            // * Load active expert i's token j's partial data
            // Offset within (num_act_exp, num_token, hidden_size) in unit of element
            int thread_offset_across_actexp_token = actexp_i * (hidden_size * num_token) + thread_offset_across_token;
            ACC_TYPE actexp_i_data;
            actexp_i_data.packed = reinterpret_cast<int4 const*>(
                ggemm2_actexp_m_hidden_in)[thread_offset_across_actexp_token / ACC_TYPE::ELEM_PER_ACCESS];

            // * Load active expert i's token j's scale
            int gloabl_exp_id = actexpi_to_global_expid[actexp_i];
            int thread_offset_scale = gloabl_exp_id * num_token + token_id;
            ScaleType actexp_i_token_j_scale
                = reinterpret_cast<ScaleType const*>(scale_actexp_m_in)[thread_offset_scale];

// * acc += scale(data)
#pragma unroll
            for (int i = 0; i < ACC_TYPE::ELEM_PER_ACCESS; ++i)
            {
                // assume computation is done in ScaleType
                accumulator.unpacked[i] += static_cast<IOType>(
                    (static_cast<ScaleType>(actexp_i_data.unpacked[i]) * actexp_i_token_j_scale));
            }
        }

        // * FC2 + reduced(gGEMM2)
        ACC_TYPE fc2_data;
        fc2_data.packed
            = reinterpret_cast<int4 const*>(fc2_m_hidden_in)[thread_offset_across_token / ACC_TYPE::ELEM_PER_ACCESS];
#pragma unroll
        for (int i = 0; i < ACC_TYPE::ELEM_PER_ACCESS; ++i)
        {
            accumulator.unpacked[i] += fc2_data.unpacked[i];
        }

        // * Store
        // Only store valid section of ACC_TYPE::ELEM_PER_ACCESS
        reinterpret_cast<int4*>(reduce_m_hidden_ou)[thread_offset_across_token / ACC_TYPE::ELEM_PER_ACCESS]
            = accumulator.packed;
    }

    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename IOType, typename ScaleType>
void moe_reduction_kernel_launcher(IOType const* ggemm2_actexp_m_hidden_in, IOType const* fc2_m_hidden_in,
    ScaleType const* scale_actexp_m_in, int const* actexpi_to_global_expid, IOType* reduce_m_hidden_ou, int num_act_exp,
    int num_token, int hidden_size)
{
    // * Device Property & SM
    int device_id;
    TLLM_CUDA_CHECK(hipGetDevice(&device_id));
    hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    int sm_count = device_prop.multiProcessorCount;

    hipStream_t stream;
    hipStreamCreate(&stream);

    using ACC_TYPE = ACCESS_TYPE<IOType>;

    // * Check for launch assumption
    if (hidden_size % ACC_TYPE::ELEM_PER_ACCESS != 0)
    {
        printf("FAILED. Unable to launch as hidden_size must be multiplier of ACC_TYPE::ELEM_PER_ACCESS\n");
        return;
    }

    // * Heuristic for launch config
    // targeting low latency inference to fully utilize as much SM as possible
    int num_thread_per_token = hidden_size / ACC_TYPE::ELEM_PER_ACCESS;
    int num_warp_per_token = (num_thread_per_token + 32 - 1) / 32;
    int cluster_dim = 8;
    while (num_warp_per_token % cluster_dim != 0)
    {
        cluster_dim /= 2;
    }
    int block_dim = num_warp_per_token / cluster_dim * 32;
    int grid_dim = min(sm_count, num_token * cluster_dim) / cluster_dim * cluster_dim;

    printf(
        "* num_act_exp %d, num_token %d, hidden_size %d, num_warp_per_token %d, heuristic pick grid %d cluster %d "
        "block %d\n",
        num_act_exp, num_token, hidden_size, num_warp_per_token, grid_dim, cluster_dim, block_dim);

    // * Launch Config
    hipLaunchConfig_t config = {0};
    hipLaunchAttribute attribute[2];
    attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attribute[0].val.programmaticStreamSerializationAllowed = 1;
    attribute[1].id = cudaLaunchAttributeClusterDimension;
    attribute[1].val.clusterDim.x = cluster_dim;
    attribute[1].val.clusterDim.y = 1;
    attribute[1].val.clusterDim.z = 1;
    config.gridDim = grid_dim;
    config.blockDim = block_dim;
    config.stream = stream;
    config.numAttrs = 2;
    config.attrs = attribute;
    config.dynamicSmemBytes = 0;

    TLLM_CUDA_CHECK(
        cudaLaunchKernelEx(&config, moe_reduction_kernel<IOType, ScaleType>, ggemm2_actexp_m_hidden_in, fc2_m_hidden_in,
            scale_actexp_m_in, actexpi_to_global_expid, reduce_m_hidden_ou, num_act_exp, num_token, hidden_size));
    TLLM_CUDA_CHECK(hipPeekAtLastError());
    TLLM_CUDA_CHECK(hipDeviceSynchronize());
}

template <typename DType>
class MoEARFuseTestRunner
{
    static_assert(std::is_same_v<DType, half> || std::is_same_v<DType, __hip_bfloat16>);
    static constexpr ncclDataType_t kNCCLDataType = std::is_same_v<DType, half> ? ncclFloat16 : ncclBfloat16;
    static constexpr nvinfer1::DataType kTRTDataType
        = std::is_same_v<DType, half> ? nvinfer1::DataType::kHALF : nvinfer1::DataType::kBF16;

public:
    MoEARFuseTestRunner(int max_token_num, int hidden_dim, int max_expert_num)
        : m_mpi_comm(mpi::MpiComm::world())
    {
        m_message_size = max_token_num * hidden_dim;
        m_world_size = m_mpi_comm.getSize();
        m_rank = m_mpi_comm.getRank();
        TLLM_CUDA_CHECK(hipSetDevice(m_rank));
        ncclUniqueId id;
        if (m_rank == 0)
        {
            TLLM_NCCL_CHECK(ncclGetUniqueId(&id));
        }
        m_mpi_comm.bcast(&id, sizeof(id), mpi::MpiType::kBYTE, 0);
        TLLM_NCCL_CHECK(ncclCommInitRank(&m_nccl_comm, m_world_size, id, m_rank));

        m_allreduce_in.allocate(m_message_size * sizeof(DType));
        m_residual_in.allocate(m_message_size * sizeof(DType));
        m_residual_out.allocate(m_message_size * sizeof(DType));
        m_norm_out.allocate(m_message_size * sizeof(DType));
        m_quant_out.allocate(m_message_size * sizeof(DType));
        m_scale_out.allocate(m_message_size * sizeof(DType));
        m_rms_gamma.allocate(hidden_dim * sizeof(DType));
        m_scale_factor.allocate(sizeof(float));
        m_stream = std::make_shared<tr::CudaStream>();
        m_workspace = std::make_shared<ar_fusion::Workspace>(m_rank, m_world_size, max_token_num, hidden_dim, m_stream);

        m_params.nranks = m_world_size;
        m_params.rank = m_rank;
        m_params.dtype = kTRTDataType;
        m_params.workspace = m_workspace->get_workspace();
        m_params.allreduce_in = m_allreduce_in.device_data();
        m_params.residual_in = m_residual_in.device_data();
        m_params.residual_out = m_residual_out.device_data();
        m_params.norm_out = m_norm_out.device_data();
        m_params.quant_out = m_quant_out.device_data();
        m_params.scale_out = m_scale_out.device_data();
        m_params.rms_gamma = m_rms_gamma.device_data();
        m_params.scale_factor = m_scale_factor.device_data<float>();
        m_params.rms_eps = 1e-3;
        m_params.stream = m_stream->get();

        // * moe reduction related param
        m_max_expert_num = max_expert_num;

        // [device_num_expert, m]
        m_moe_reduction_scale_input.allocate(m_max_expert_num * max_token_num * sizeof(float));
        // [device_num_expert, m, 7168]
        m_moe_reduction_active_experts_token_input.allocate(m_max_expert_num * m_message_size * sizeof(DType));
        // [m, 7168]
        m_moe_reduction_token_input.allocate(m_message_size * sizeof(DType));
        // [1]
        m_moe_reduction_device_num_experts.allocate(sizeof(int));

        m_params.moe_reduction_scale_input = reinterpret_cast<float*>(m_moe_reduction_scale_input.device_data());
        m_params.moe_reduction_active_experts_token_input = m_moe_reduction_active_experts_token_input.device_data();
        m_params.moe_reduction_token_input = m_moe_reduction_token_input.device_data();
        m_params.moe_reduction_device_num_experts
            = reinterpret_cast<int*>(m_moe_reduction_device_num_experts.device_data());
    }

    void random_input()
    {
        m_allreduce_in.random<DType>(-100.f, 100.f);
        m_residual_in.random<DType>(-100.f, 100.f);
        m_rms_gamma.random<DType>(-1.f, 1.f);
        m_scale_factor.random<float>(5.f, 5.f);

        // * moe reduction
        m_moe_reduction_scale_input.random<float>(-100.f, 100.f);
        m_moe_reduction_active_experts_token_input.random<DType>(-100.f, 100.f);
        m_moe_reduction_token_input.random<DType>(-100.f, 100.f);
    }

    template <typename Func>
    float benchmark(Func func, int warmup, int iter, int token_num, int hidden_dim, int num_active_expert = 0)
    {
        m_params.size = token_num * hidden_dim;
        m_params.hidden_dim = hidden_dim;
        hipMemcpy(m_params.moe_reduction_device_num_experts, &num_active_expert, sizeof(int), hipMemcpyHostToDevice);
        hipEvent_t begin, end;
        hipEventCreate(&begin);
        hipEventCreate(&end);
        random_input();
        m_mpi_comm.barrier();
        for (int i = 0; i < warmup; ++i)
        {
            (this->*func)(token_num, hidden_dim, num_active_expert);
        }
        hipEventRecord(begin, m_stream->get());
        for (int i = 0; i < iter; ++i)
        {
            (this->*func)(token_num, hidden_dim, num_active_expert);
        }
        hipEventRecord(end, m_stream->get());
        hipEventSynchronize(end);
        float time;
        hipEventElapsedTime(&time, begin, end);
        time /= iter;
        m_mpi_comm.barrier();
        hipEventDestroy(begin);
        hipEventDestroy(end);
        return time * 1000;
    }

    int get_sm_count() const
    {
        static int sm_count = 0;
        if (sm_count == 0)
        {
            int device_id;
            TLLM_CUDA_CHECK(hipGetDevice(&device_id));
            hipDeviceProp_t device_prop;
            hipGetDeviceProperties(&device_prop, device_id);
            sm_count = device_prop.multiProcessorCount;
        }
        return sm_count;
    }

    void verify(int token_num, int hidden_dim, int num_active_expert)
    {
        int message_size = token_num * hidden_dim;
        CudaBuffer ref_output(message_size * sizeof(DType)), ref_scale(message_size * sizeof(DType));

        // * MoE Reduction
        moe_reduction_kernel_launcher<DType, float>(m_moe_reduction_active_experts_token_input.device_data<DType>(),
            m_moe_reduction_token_input.device_data<DType>(), m_moe_reduction_scale_input.device_data<float>(),
            ref_output.device_data<DType>(), num_active_expert, token_num, hidden_dim);

        compare<DType>(
            m_rank, m_allreduce_in.host_data(), ref_output.host_data(), message_size, 1e-3, "moe reduction out");

        // * AR
        TLLM_NCCL_CHECK(ncclAllReduce(m_allreduce_in.device_data(), ref_output.device_data(), message_size,
            kNCCLDataType, ncclSum, m_nccl_comm, 0));

        // * Add
        residual_add(ref_output.device_data<DType>(), m_residual_in.device_data<DType>(), message_size, 0);

        // * Norm
        invokeGeneralRmsNorm<DType, int8_t>(ref_output.device_data<DType>(), ref_output.device_data<DType>(),
            m_rms_gamma.device_data<DType>(), nullptr, m_params.rms_eps, token_num, hidden_dim,
            tensorrt_llm::common::QuantMode(), 0);

        compare<DType>(m_rank, m_norm_out.host_data(), ref_output.host_data(), message_size, 1e-3, "norm out");

        // * Quant
        invokeFP4Quantization(token_num, hidden_dim, m_norm_out.device_data<DType>(),
            m_scale_factor.device_data<float>(), ref_output.device_data<int64_t>(), ref_scale.device_data<int32_t>(),
            false, tensorrt_llm::FP4QuantizationSFLayout::SWIZZLED, 128, 0);
        compare<int8_t>(m_rank, m_quant_out.host_data(), ref_output.host_data(), message_size / 2, 1e-3, "quant out");
        compare<int8_t>(m_rank, m_scale_out.host_data(), ref_scale.host_data(), message_size / 16, 1e-3, "scale out");
    }

    void run_nccl_allreduce(int token_num, int hidden_dim, int)
    {
        TLLM_NCCL_CHECK(ncclAllReduce(m_allreduce_in.device_data(), m_residual_out.device_data(),
            token_num * hidden_dim, kNCCLDataType, ncclSum, m_nccl_comm, m_stream->get()));
    }

    void run_moe_reduction(int token_num, int hidden_dim, int num_active_expert)
    {
        moe_reduction_kernel_launcher<DType, float>(m_moe_reduction_active_experts_token_input.device_data<DType>(),
            m_moe_reduction_token_input.device_data<DType>(), m_moe_reduction_scale_input.device_data<float>(),
            m_allreduce_in.device_data<DType>(), num_active_expert, token_num, hidden_dim);
    }

    void run_residual_add(int token_num, int hidden_dim, int)
    {
        residual_add(m_residual_out.device_data<DType>(), // output and input
            m_residual_in.device_data<DType>(),           // input
            token_num * hidden_dim, m_stream->get());
    }

    void run_rms_norm(int token_num, int hidden_dim, int)
    {
        invokeGeneralRmsNorm<DType, int8_t>(m_residual_out.device_data<DType>(), m_norm_out.device_data<DType>(),
            m_rms_gamma.device_data<DType>(), nullptr, m_params.rms_eps, token_num, hidden_dim,
            tensorrt_llm::common::QuantMode(), m_stream->get());
    }

    void run_fp4_quant(int token_num, int hidden_dim, int)
    {
        invokeFP4Quantization(token_num,         // m
            hidden_dim,                          // n
            m_norm_out.device_data<DType>(),     // input
            m_scale_factor.device_data<float>(), // input sf
            m_quant_out.device_data<int64_t>(),  // output
            m_scale_out.device_data<int32_t>(),  // output sf
            false, tensorrt_llm::FP4QuantizationSFLayout::SWIZZLED, 128, m_stream->get());
    }

    void run_kernel(int token_num, int hidden_dim)
    {
        ar_fusion::moe::moereduction_allreduce_fusion_op(m_params);
    }

    ~MoEARFuseTestRunner()
    {
        TLLM_NCCL_CHECK(ncclCommDestroy(m_nccl_comm));
    }

private:
    int m_rank;
    int m_world_size;
    int m_message_size;
    mpi::MpiComm const& m_mpi_comm;
    ncclComm_t m_nccl_comm;
    CudaBuffer m_allreduce_in;
    CudaBuffer m_residual_in;
    CudaBuffer m_residual_out;
    CudaBuffer m_norm_out;
    CudaBuffer m_quant_out;
    CudaBuffer m_scale_out;
    CudaBuffer m_rms_gamma;
    CudaBuffer m_scale_factor;
    std::shared_ptr<ar_fusion::Workspace> m_workspace;
    ar_fusion::moe::MoeReductionAllReduceFusionParams m_params;
    std::shared_ptr<tr::CudaStream> m_stream;

    // * moe reduction related params
    int m_max_expert_num;
    CudaBuffer m_moe_reduction_scale_input;
    CudaBuffer m_moe_reduction_active_experts_token_input;
    CudaBuffer m_moe_reduction_token_input;
    CudaBuffer m_moe_reduction_device_num_experts;
};

TEST(Kernel, MoEReduceAddARFuse)
{
    auto& comm = mpi::MpiComm::world();
    auto world_size = comm.getSize();
    auto rank = comm.getRank();
    if (world_size % 2)
    {
        TLLM_LOG_WARNING("world size is not a multiple of 2, return");
        return;
    }
    int warmup = 100, iter = 100;
    int hidden_dim = 7168;
    std::vector<int> candidate_token_num{1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 1024, 2048};
    std::vector<int> candidate_active_expert_num{8, 12, 16};
    int max_token_num = 2048;
    int max_expert_num = 16;
    MoEARFuseTestRunner<half> runner(max_token_num, hidden_dim, max_expert_num);
    for (auto token_num : candidate_token_num)
    {
        for (auto act_exp_num : candidate_active_expert_num)
        {
            auto latency = runner.benchmark(
                &MoEARFuseTestRunner<half>::run_kernel, warmup, iter, token_num, hidden_dim, act_exp_num);
            runner.verify(token_num, hidden_dim, act_exp_num);
            if (rank == 0)
            {
                TLLM_LOG_INFO("token_num %d, hidden_dim %d, act_exp_num %d, latency %fus", token_num, hidden_dim,
                    act_exp_num, latency);
            }
            auto moe_reduce_latency = runner.benchmark(
                &MoEARFuseTestRunner<half>::run_moe_reduction, warmup, iter, token_num, hidden_dim, act_exp_num);
            if (rank == 0)
            {
                TLLM_LOG_INFO("moe reduce latency %fus", moe_reduce_latency);
            }
            auto nccl_latency
                = runner.benchmark(&MoEARFuseTestRunner<half>::run_nccl_allreduce, warmup, iter, token_num, hidden_dim);
            if (rank == 0)
            {
                TLLM_LOG_INFO("nccl allreduce latency %fus", nccl_latency);
            }
            auto residual_latency
                = runner.benchmark(&MoEARFuseTestRunner<half>::run_residual_add, warmup, iter, token_num, hidden_dim);
            if (rank == 0)
            {
                TLLM_LOG_INFO("residual add latency %fus", residual_latency);
            }
            auto rms_latency
                = runner.benchmark(&MoEARFuseTestRunner<half>::run_rms_norm, warmup, iter, token_num, hidden_dim);
            if (rank == 0)
            {
                TLLM_LOG_INFO("rms norm latency %fus", rms_latency);
            }
            auto quant_latency
                = runner.benchmark(&MoEARFuseTestRunner<half>::run_fp4_quant, warmup, iter, token_num, hidden_dim);
            if (rank == 0)
            {
                TLLM_LOG_INFO("fp4 quant latency %fus", quant_latency);
                auto tot_latency = moe_reduce_latency + nccl_latency + residual_latency + rms_latency + quant_latency;
                TLLM_LOG_INFO("fusion kernel latency %fus, moe reduce + nccl + ops latency %fus, total speedup %fx",
                    latency, tot_latency, tot_latency / latency);
            }
        }
    }
}
