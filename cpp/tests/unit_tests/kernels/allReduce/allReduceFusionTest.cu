#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime.h>
#include <gtest/gtest.h>
#include <nccl.h>

#include <cstdarg>
#include <cstdint>
#include <functional>
#include <iostream>
#include <random>
#include <vector>

#include "tensorrt_llm/kernels/communicationKernels/allReduceFusionKernels.h"
#include "tensorrt_llm/kernels/communicationKernels/allReduceWorkspace.h"
#include "tensorrt_llm/kernels/quantization.h"
#include "tensorrt_llm/kernels/rmsnormKernels.h"
#include "tensorrt_llm/runtime/cudaStream.h"
#include "tensorrt_llm/runtime/utils/mpiUtils.h"
#include "tensorrt_llm/runtime/utils/multiDeviceUtils.h"

namespace mpi = tensorrt_llm::mpi;
namespace tr = tensorrt_llm::runtime;
using namespace tensorrt_llm::kernels;

template <typename DType>
__global__ void residual_add_kernel(DType* data, DType* residual, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;
    data[idx] = data[idx] + residual[idx];
}

template <typename DType>
void residual_add(DType* data, DType* residual, int size, hipStream_t stream)
{
    residual_add_kernel<<<size / 128, 128, 0, stream>>>(data, residual, size);
}

template <typename DType>
__global__ void quantize_to_fp8_kernel(DType* data, __hip_fp8_e4m3_fnuz* data_fp8, int size, float* scale_factor)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size)
        return;
    data_fp8[idx] = static_cast<__hip_fp8_e4m3_fnuz>(static_cast<float>(data[idx]) * (1.f / *scale_factor));
}

template <typename DType>
void quantize_to_fp8(DType* data, __hip_fp8_e4m3_fnuz* data_fp8, int size, float* scale_factor, hipStream_t stream)
{
    quantize_to_fp8_kernel<<<size / 128, 128, 0, stream>>>(data, data_fp8, size, scale_factor);
}

template <typename T>
bool compare(int rank, void* p_real, void* p_ref, int size, std::string const& cmp_info = "", float atol = 1e-3)
{
    auto ptr_real = reinterpret_cast<T*>(p_real);
    auto ptr_ref = reinterpret_cast<T*>(p_ref);
    float max_diff = 0.f, tot_diff = 0.f;
    int error_cnt = 0;
    float max_error_value_real = 0.f, max_error_value_ref = 0.f;
    static char* ar_debug = std::getenv("AR_DEBUG");
    if (ar_debug && rank == 0)
    {
        printf("TensorReal: [");
        for (int n = 0; n < 20; ++n)
        {
            float v = static_cast<float>(ptr_real[n]);
            printf("%f, ", v);
        }
        printf("...]\n");
        printf("TensorRef: [");
        for (int n = 0; n < 20; ++n)
        {
            float v = static_cast<float>(ptr_ref[n]);
            printf("%f, ", v);
        }
        printf("...]\n");
    }
    int print_cnt = 0;
    for (int n = 0; n < size; ++n)
    {
        float v_real = static_cast<float>(ptr_real[n]);
        float v_ref = static_cast<float>(ptr_ref[n]);
        float diff = std::abs(v_real - v_ref);

        if (diff > max_diff)
        {
            max_diff = diff;
            max_error_value_real = v_real;
            max_error_value_ref = v_ref;
        }

        bool is_error = diff > atol;
        if (diff > atol)
        {
            tot_diff += diff;
            ++error_cnt;
        }
        if (ar_debug && is_error && rank == 0 && print_cnt < 20)
        {
            ++print_cnt;
            if (rank == 0)
                printf("idx %d, v_real %f, v_ref %f\n", n, v_real, v_ref);
        }
    }
    bool pass = error_cnt == 0;
    if (!pass && rank == 0)
    {
        printf(
            "[%s] rank %d, atol %8.4f, max absolute diff %8.4f(%8.4f vs %8.4f), avg absolute diff %8.4f, absolute "
            "error count %d/%d\n",
            cmp_info.c_str(), rank, atol, max_diff, max_error_value_real, max_error_value_ref,
            tot_diff / std::max(error_cnt, 1), error_cnt, size);
    }
    return pass;
}

template <typename T1, typename T2>
void random_fill(T1* data, int size, T2 minv, T2 maxv)
{
    static int rseed = 20250227;
    std::mt19937 gen(rseed++);
    std::uniform_real_distribution<float> dis(static_cast<float>(minv), static_cast<float>(maxv));
    for (int i = 0; i < size; ++i)
    {
        data[i] = static_cast<T1>(dis(gen));
    }
}

int get_random_int(int min_v, int max_v)
{
    static int rseed = 20250227;
    std::mt19937 gen(rseed++);
    std::uniform_int_distribution<> dis(min_v, max_v);

    return dis(gen);
}

struct CudaBuffer
{
    void* m_d_data;
    void* m_h_data;
    int m_size;

    CudaBuffer(int size_in_bytes = 0)
        : m_size(size_in_bytes)
        , m_d_data(nullptr)
        , m_h_data(nullptr)
    {
        allocate(size_in_bytes);
    }

    void allocate(int size_in_bytes)
    {
        if (size_in_bytes == 0)
            return;
        TLLM_CHECK(m_d_data == nullptr && m_h_data == nullptr);
        m_size = size_in_bytes;
        TLLM_CUDA_CHECK(hipMalloc(&m_d_data, m_size));
        clear();
        m_h_data = malloc(m_size);
    }

    template <typename T = void>
    T* device_data()
    {
        TLLM_CHECK(m_d_data != nullptr);
        return reinterpret_cast<T*>(m_d_data);
    }

    template <typename T = void>
    T* host_data()
    {
        TLLM_CHECK(m_h_data != nullptr);
        d2h();
        return reinterpret_cast<T*>(m_h_data);
    }

    template <typename DType, typename VType>
    void random(VType minv, VType maxv)
    {
        random_fill(reinterpret_cast<DType*>(m_h_data), m_size / sizeof(DType), minv, maxv);
        h2d();
    }

    void clear()
    {
        TLLM_CUDA_CHECK(hipMemset(m_d_data, 0, m_size));
    }

    void h2d()
    {
        TLLM_CUDA_CHECK(hipMemcpy(m_d_data, m_h_data, m_size, hipMemcpyHostToDevice));
    }

    void d2h()
    {
        TLLM_CUDA_CHECK(hipMemcpy(m_h_data, m_d_data, m_size, hipMemcpyDeviceToHost));
    }

    ~CudaBuffer()
    {
        if (m_d_data)
        {
            TLLM_CUDA_CHECK(hipFree(m_d_data));
        }
        if (m_h_data)
        {
            free(m_h_data);
        }
    }
};

template <typename DType>
struct DTypeTraits;

template <>
struct DTypeTraits<half>
{
    static constexpr ncclDataType_t kNCCLDataType = ncclFloat16;
    static constexpr nvinfer1::DataType kTRTDataType = nvinfer1::DataType::kHALF;
};

template <>
struct DTypeTraits<__hip_bfloat16>
{
    static constexpr ncclDataType_t kNCCLDataType = ncclBfloat16;
    static constexpr nvinfer1::DataType kTRTDataType = nvinfer1::DataType::kBF16;
};

template <>
struct DTypeTraits<float>
{
    static constexpr ncclDataType_t kNCCLDataType = ncclFloat32;
    static constexpr nvinfer1::DataType kTRTDataType = nvinfer1::DataType::kFLOAT;
};

template <typename DType, ar_fusion::AllReduceFusionPattern Pattern>
class TestRunner
{
    static constexpr ncclDataType_t kNCCLDataType = DTypeTraits<DType>::kNCCLDataType;
    static constexpr nvinfer1::DataType kTRTDataType = DTypeTraits<DType>::kTRTDataType;
    static constexpr bool kFP4QuantOutSupport = !std::is_same_v<DType, float>;
    static_assert(kFP4QuantOutSupport || Pattern != ar_fusion::AllReduceFusionPattern::kARResidualRMSNormFP4Quant,
        "kARResidualRMSNormFP4Quant is not supported for float dtype");

public:
    TestRunner(int max_token_num, int hidden_dim)
        : m_mpi_comm(mpi::MpiComm::world())
    {
        m_message_size = max_token_num * hidden_dim;
        m_world_size = m_mpi_comm.getSize();
        m_rank = m_mpi_comm.getRank();
        TLLM_CUDA_CHECK(hipSetDevice(m_rank));
        ncclUniqueId id;
        if (m_rank == 0)
        {
            TLLM_NCCL_CHECK(ncclGetUniqueId(&id));
        }
        m_mpi_comm.bcast(&id, sizeof(id), mpi::MpiType::kBYTE, 0);
        TLLM_NCCL_CHECK(ncclCommInitRank(&m_nccl_comm, m_world_size, id, m_rank));

        m_allreduce_in.allocate(m_message_size * sizeof(DType));
        m_residual_in.allocate(m_message_size * sizeof(DType));
        m_allreduce_out.allocate(m_message_size * sizeof(DType));
        m_residual_out.allocate(m_message_size * sizeof(DType));
        m_norm_out.allocate(m_message_size * sizeof(DType));
        m_quant_out.allocate(m_message_size * sizeof(DType));
        // SF layout was packed to [numMTiles, numKTiles, 32 (mTile), 4 (mTile), 4(kTile)]
        size_t scale_out_size = ((max_token_num + 127) / 128 * 128) * ((hidden_dim + 63) / 64 * 4);
        m_scale_out.allocate(scale_out_size);
        m_rms_gamma.allocate(hidden_dim * sizeof(DType));
        m_scale_factor.allocate(sizeof(float));
        m_stream = std::make_shared<tr::CudaStream>();
        m_workspace = std::make_shared<ar_fusion::Workspace>(m_rank, m_world_size, max_token_num, hidden_dim, m_stream);

        m_params.nranks = m_world_size;
        m_params.rank = m_rank;
        m_params.dtype = kTRTDataType;
        m_params.workspace = m_workspace->get_workspace();
        m_params.allreduce_in = m_allreduce_in.device_data();
        m_params.residual_in = m_residual_in.device_data();
        m_params.allreduce_out = m_allreduce_out.device_data();
        m_params.residual_out = m_residual_out.device_data();
        m_params.norm_out = m_norm_out.device_data();
        m_params.quant_out = m_quant_out.device_data();
        m_params.scale_out = m_scale_out.device_data();
        m_params.rms_gamma = m_rms_gamma.device_data();
        m_params.scale_factor = m_scale_factor.device_data<float>();
        m_params.rms_eps = 1e-3;
        m_params.stream = m_stream->get();
        m_params.pattern = Pattern;
    }

    void reset_io()
    {
        m_allreduce_in.random<DType>(-100.f, 100.f);
        m_residual_in.random<DType>(-100.f, 100.f);
        m_rms_gamma.random<DType>(-1.f, 1.f);
        m_scale_factor.random<float>(1.f, 1.f);
        // Because scale_out internally performs layout interleaving, not all elements will be covered, so it should be
        // reset before calling the kernel to ensure correct comparison results
        if (kFP4QuantOutSupport)
        {
            m_scale_out.clear();
        }
    }

    template <typename Func>
    float benchmark(Func func, int warmup, int iter, int token_num, int hidden_dim)
    {
        m_params.size = token_num * hidden_dim;
        m_params.hidden_dim = hidden_dim;
        hipEvent_t begin, end;
        hipEventCreate(&begin);
        hipEventCreate(&end);
        m_mpi_comm.barrier();
        for (int i = 0; i < warmup; ++i)
        {
            (this->*func)(token_num, hidden_dim);
        }
        hipEventRecord(begin, m_stream->get());
        for (int i = 0; i < iter; ++i)
        {
            (this->*func)(token_num, hidden_dim);
        }
        hipEventRecord(end, m_stream->get());
        hipEventSynchronize(end);
        float time;
        hipEventElapsedTime(&time, begin, end);
        time /= iter;
        m_mpi_comm.barrier();
        hipEventDestroy(begin);
        hipEventDestroy(end);
        return time * 1000;
    }

    template <typename Func>
    void run_once(Func func, int token_num, int hidden_dim)
    {
        benchmark(func, 0, 1, token_num, hidden_dim);
    }

    int get_sm_count()
    {
        static int sm_count = 0;
        if (sm_count == 0)
        {
            int device_id;
            TLLM_CUDA_CHECK(hipGetDevice(&device_id));
            hipDeviceProp_t device_prop;
            hipGetDeviceProperties(&device_prop, device_id);
            sm_count = device_prop.multiProcessorCount;
        }
        return sm_count;
    }

    void verify(int token_num, int hidden_dim)
    {
        int message_size = token_num * hidden_dim;
        CudaBuffer ref_output(message_size * sizeof(DType));

        // We directly compare the results of AR+AddResidual here, as the accumulation order in NCCL's AR might be
        // inconsistent across different kernels. Therefore, we set atol to 1 (setting it to 0 locally also passes the
        // test).
        TLLM_NCCL_CHECK(ncclAllReduce(m_allreduce_in.device_data(), ref_output.device_data(), message_size,
            kNCCLDataType, ncclSum, m_nccl_comm, 0));
        if constexpr (ar_fusion::HasAllReduceOut<Pattern>)
        {
            TLLM_CHECK(compare<DType>(
                m_rank, m_allreduce_out.host_data(), ref_output.host_data(), message_size, "allreduce out", 1));
        }
        if constexpr (ar_fusion::HasResidual<Pattern>)
        {
            residual_add(ref_output.device_data<DType>(), m_residual_in.device_data<DType>(), message_size, 0);
            if constexpr (ar_fusion::HasResidualOut<Pattern>)
            {
                TLLM_CHECK(compare<DType>(
                    m_rank, m_residual_out.host_data(), ref_output.host_data(), message_size, "residual out", 1));
            }
        }
        if constexpr (ar_fusion::HasRMSNorm<Pattern>)
        {
            // This excludes the accumulation order errors introduced by AR and only compares the accuracy of the
            // RMSNorm. The atol is set to 1e-2 to exclude errors caused by accumulation order changes due to
            // differences in cluster/block size.
            invokeGeneralRmsNorm<DType, int8_t>(ref_output.device_data<DType>(), m_residual_out.device_data<DType>(),
                m_rms_gamma.device_data<DType>(), nullptr, m_params.rms_eps, token_num, hidden_dim,
                tensorrt_llm::common::QuantMode(), 0);
            if constexpr (ar_fusion::HasNormOut<Pattern>)
            {
                TLLM_CHECK(compare<DType>(
                    m_rank, m_norm_out.host_data(), ref_output.host_data(), message_size, "norm out", 1e-2));
            }
        }
        if constexpr (ar_fusion::GetQuantType<Pattern> == ar_fusion::QuantType::kFP4)
        {
            // We need norm out to verify the accuracy of quantization.
            static_assert(Pattern == ar_fusion::AllReduceFusionPattern::kARResidualRMSNormOutFP4Quant);
            // SF layout was packed to [numMTiles, numKTiles, 32 (mTile), 4 (mTile), 4(kTile)]
            size_t scale_out_size = ((token_num + 127) / 128 * 128) * ((hidden_dim + 63) / 64 * 4);
            CudaBuffer ref_scale(scale_out_size);
            // Here, we also only compare the accuracy of quantization. Since there are no differences in
            // computation order, atol is set to 0.
            invokeFP4Quantization(token_num, hidden_dim, m_norm_out.device_data<DType>(),
                m_scale_factor.device_data<float>(), ref_output.device_data<int64_t>(),
                ref_scale.device_data<int32_t>(), false, tensorrt_llm::FP4QuantizationSFLayout::SWIZZLED, 128, 0);
            TLLM_CHECK(compare<int8_t>(
                m_rank, m_quant_out.host_data(), ref_output.host_data(), message_size / 2, "fp4 quant out", 0));
            TLLM_CHECK(compare<int8_t>(
                m_rank, m_scale_out.host_data(), ref_scale.host_data(), scale_out_size, "fp4 scale out", 0));
        }
        else if constexpr (ar_fusion::GetQuantType<Pattern> == ar_fusion::QuantType::kFP8)
        {
            // We need norm out to verify the accuracy of quantization.
            static_assert(Pattern == ar_fusion::AllReduceFusionPattern::kARResidualRMSNormOutFP8Quant);
            CudaBuffer ref_fp8_output(message_size * sizeof(__hip_fp8_e4m3_fnuz));
            quantize_to_fp8(m_norm_out.device_data<DType>(), ref_fp8_output.device_data<__hip_fp8_e4m3_fnuz>(), message_size,
                m_scale_factor.device_data<float>(), m_stream->get());
            TLLM_CHECK(compare<__hip_fp8_e4m3_fnuz>(
                m_rank, m_quant_out.host_data(), ref_fp8_output.host_data(), message_size, "fp8 quant out", 0));
        }
    }

    void run_nccl_allreduce(int token_num, int hidden_dim)
    {
        TLLM_NCCL_CHECK(ncclAllReduce(m_allreduce_in.device_data(), m_residual_out.device_data(),
            token_num * hidden_dim, kNCCLDataType, ncclSum, m_nccl_comm, m_stream->get()));
    }

    void run_residual_add(int token_num, int hidden_dim)
    {
        residual_add(m_residual_out.device_data<DType>(), m_residual_in.device_data<DType>(), token_num * hidden_dim,
            m_stream->get());
    }

    void run_rms_norm(int token_num, int hidden_dim)
    {
        invokeGeneralRmsNorm<DType, int8_t>(m_norm_out.device_data<DType>(), m_residual_out.device_data<DType>(),
            m_rms_gamma.device_data<DType>(), nullptr, m_params.rms_eps, token_num, hidden_dim,
            tensorrt_llm::common::QuantMode(), m_stream->get());
    }

    void run_fp4_quant(int token_num, int hidden_dim)
    {
        invokeFP4Quantization(token_num, hidden_dim, m_norm_out.device_data<DType>(),
            m_scale_factor.device_data<float>(), m_quant_out.device_data<int64_t>(), m_scale_out.device_data<int32_t>(),
            false, tensorrt_llm::FP4QuantizationSFLayout::SWIZZLED, 128, m_stream->get());
    }

    void run_kernel(int token_num, int hidden_dim)
    {
        ar_fusion::allreduce_fusion_op(m_params);
    }

    ~TestRunner()
    {
        TLLM_NCCL_CHECK(ncclCommDestroy(m_nccl_comm));
    }

private:
    int m_rank;
    int m_world_size;
    int m_message_size;
    mpi::MpiComm const& m_mpi_comm;
    ncclComm_t m_nccl_comm;
    CudaBuffer m_allreduce_in;
    CudaBuffer m_residual_in;
    CudaBuffer m_allreduce_out;
    CudaBuffer m_residual_out;
    CudaBuffer m_norm_out;
    CudaBuffer m_quant_out;
    CudaBuffer m_scale_out;
    CudaBuffer m_rms_gamma;
    CudaBuffer m_scale_factor;
    std::shared_ptr<ar_fusion::Workspace> m_workspace;
    ar_fusion::AllReduceFusionParams m_params;
    std::shared_ptr<tr::CudaStream> m_stream;
};

TEST(Kernel_AllReduceFusion, AllReduceAccuracyRandomTokenNum)
{
    using Runner = TestRunner<half, ar_fusion::AllReduceFusionPattern::kAllReduce>;
    auto& comm = mpi::MpiComm::world();
    auto world_size = comm.getSize();
    auto rank = comm.getRank();
    if (world_size % 2)
    {
        TLLM_LOG_WARNING("world size is not a multiple of 2, return");
        return;
    }
    int iter = 100;
    std::vector<int> candidate_hidden_dim{1024, 2048, 4096, 7168, 8192};
    int min_token_num = 1;
    int max_token_num = 2048;
    for (auto hidden_dim : candidate_hidden_dim)
    {
        Runner runner(max_token_num, hidden_dim);
        for (int i = 0; i < iter; ++i)
        {
            int token_num = get_random_int(min_token_num, max_token_num);
            if (rank == 0)
            {
                printf("[Verify] token_num %-4d, hidden_dim %-4d ...", token_num, hidden_dim);
            }
            runner.reset_io();
            runner.run_once(&Runner::run_kernel, token_num, hidden_dim);
            runner.verify(token_num, hidden_dim);
            if (rank == 0)
            {
                printf("\033[32mPass!\033[0m\n");
            }
        }
    }
}

TEST(Kernel_AllReduceFusion, AllReduceAccuracyFixedTokenNum)
{
    using Runner = TestRunner<half, ar_fusion::AllReduceFusionPattern::kAllReduce>;
    auto& comm = mpi::MpiComm::world();
    auto world_size = comm.getSize();
    auto rank = comm.getRank();
    if (world_size % 2)
    {
        TLLM_LOG_WARNING("world size is not a multiple of 2, return");
        return;
    }
    int iter = 10;
    std::vector<int> candidate_hidden_dim{1024, 2048, 4096, 7168, 8192};
    int min_token_num = 1;
    int max_token_num = 2048;
    for (auto hidden_dim : candidate_hidden_dim)
    {
        Runner runner(max_token_num, hidden_dim);
        for (int token_num = min_token_num; token_num <= max_token_num; token_num *= 2)
        {
            if (rank == 0)
            {
                printf("[Verify] token_num %-4d, hidden_dim %-4d ...", token_num, hidden_dim);
            }
            for (int i = 0; i < iter; ++i)
            {
                runner.reset_io();
                runner.run_once(&Runner::run_kernel, token_num, hidden_dim);
                runner.verify(token_num, hidden_dim);
            }
            if (rank == 0)
            {
                printf("\033[32mPass!\033[0m\n");
            }
        }
    }
}

TEST(Kernel_AllReduceFusion, AllReduceFusionAccuracyDifferentDType)
{
#define TEST_AR_FUSION(DType, FusionPattern)                                                                           \
    {                                                                                                                  \
        using Runner = TestRunner<DType, FusionPattern>;                                                               \
        Runner runner(max_token_num, hidden_dim);                                                                      \
        for (int token_num = min_token_num; token_num <= max_token_num; token_num *= 2)                                \
        {                                                                                                              \
            if (rank == 0)                                                                                             \
            {                                                                                                          \
                printf("[Verify] pattern %-20s, dtype %-10s, token_num %-4d, hidden_dim %-4d ...", #FusionPattern,     \
                    #DType, token_num, hidden_dim);                                                                    \
            }                                                                                                          \
            runner.reset_io();                                                                                         \
            runner.run_once(&Runner::run_kernel, token_num, hidden_dim);                                               \
            runner.verify(token_num, hidden_dim);                                                                      \
            if (rank == 0)                                                                                             \
            {                                                                                                          \
                printf("\033[32mPass!\033[0m\n");                                                                      \
            }                                                                                                          \
        }                                                                                                              \
    }

    auto& comm = mpi::MpiComm::world();
    auto world_size = comm.getSize();
    auto rank = comm.getRank();
    if (world_size % 2)
    {
        TLLM_LOG_WARNING("world size is not a multiple of 2, return");
        return;
    }
    std::vector<int> candidate_hidden_dim{1024, 2048, 4096, 7168, 8192};
    int min_token_num = 1;
    int max_token_num = 2048;
    for (auto hidden_dim : candidate_hidden_dim)
    {
        TEST_AR_FUSION(half, ar_fusion::AllReduceFusionPattern::kAllReduce);
        TEST_AR_FUSION(__hip_bfloat16, ar_fusion::AllReduceFusionPattern::kAllReduce);
        TEST_AR_FUSION(float, ar_fusion::AllReduceFusionPattern::kAllReduce);
        TEST_AR_FUSION(half, ar_fusion::AllReduceFusionPattern::kARResidualRMSNorm);
        TEST_AR_FUSION(__hip_bfloat16, ar_fusion::AllReduceFusionPattern::kARResidualRMSNorm);
        TEST_AR_FUSION(float, ar_fusion::AllReduceFusionPattern::kARResidualRMSNorm);
        TEST_AR_FUSION(half, ar_fusion::AllReduceFusionPattern::kARResidualRMSNormOutFP8Quant);
        TEST_AR_FUSION(__hip_bfloat16, ar_fusion::AllReduceFusionPattern::kARResidualRMSNormOutFP8Quant);
        TEST_AR_FUSION(float, ar_fusion::AllReduceFusionPattern::kARResidualRMSNormOutFP8Quant);
        TEST_AR_FUSION(half, ar_fusion::AllReduceFusionPattern::kARResidualRMSNormOutFP4Quant);
        TEST_AR_FUSION(__hip_bfloat16, ar_fusion::AllReduceFusionPattern::kARResidualRMSNormOutFP4Quant);
    }
#undef TEST_AR_FUSION
}

TEST(Kernel_AllReduceFusion, Perf)
{
    using Runner = TestRunner<half, ar_fusion::AllReduceFusionPattern::kARResidualRMSNormFP4Quant>;
    auto& comm = mpi::MpiComm::world();
    auto world_size = comm.getSize();
    auto rank = comm.getRank();
    if (world_size % 2)
    {
        TLLM_LOG_WARNING("world size is not a multiple of 2, return");
        return;
    }
    int warmup = 100, iter = 300;
    int hidden_dim = 7168;
    std::vector<int> candidate_token_num{1, 2, 4, 8, 16, 32, 64, 128, 256, 512, 1024, 2048};
    int max_token_num = 2048;
    Runner runner(max_token_num, hidden_dim);
    for (auto token_num : candidate_token_num)
    {
        auto latency = runner.benchmark(&Runner::run_kernel, warmup, iter, token_num, hidden_dim);
        if (rank == 0)
        {
            TLLM_LOG_INFO(
                "token_num %-4d, hidden_dim %-4d, fusion kernel latency %4.4fus", token_num, hidden_dim, latency);
        }
        auto nccl_latency = runner.benchmark(&Runner::run_nccl_allreduce, warmup, iter, token_num, hidden_dim);
        if (rank == 0)
        {
            TLLM_LOG_INFO("nccl allreduce latency %4.4fus", nccl_latency);
        }
        auto residual_latency = runner.benchmark(&Runner::run_residual_add, warmup, iter, token_num, hidden_dim);
        if (rank == 0)
        {
            TLLM_LOG_INFO("residual add latency %4.4fus", residual_latency);
        }
        auto rms_latency = runner.benchmark(&Runner::run_rms_norm, warmup, iter, token_num, hidden_dim);
        if (rank == 0)
        {
            TLLM_LOG_INFO("rms norm latency %4.4fus", rms_latency);
        }
        auto quant_latency = runner.benchmark(&Runner::run_fp4_quant, warmup, iter, token_num, hidden_dim);
        if (rank == 0)
        {
            TLLM_LOG_INFO("fp4 quant latency %4.4fus", quant_latency);
            auto tot_latency = nccl_latency + residual_latency + rms_latency + quant_latency;
            TLLM_LOG_INFO("fusion kernel latency %4.4fus, nccl + ops latency %4.4fus, total speedup %2.4fx", latency,
                tot_latency, tot_latency / latency);
        }
    }
}
