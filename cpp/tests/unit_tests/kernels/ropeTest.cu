#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <gtest/gtest.h>

#include "tensorrt_llm/common/quantization.h"
#include "tensorrt_llm/kernels/decodingCommon.h"
#include "tensorrt_llm/kernels/kvCacheUtils.h"
#include "tensorrt_llm/kernels/unfusedAttentionKernels.h"
#include "tensorrt_llm/plugins/gptAttentionCommon/gptAttentionCommon.h"
#include "tensorrt_llm/runtime/bufferManager.h"

#include <random>
#include <tensorrt_llm/kernels/quantization.h>

#ifdef ENABLE_FP4
#include <hip/hip_fp4.h>
#endif

using namespace tensorrt_llm::runtime;
using namespace tensorrt_llm::kernels;

#ifdef ENABLE_FP4
template <typename fpType>
void computeReferenceFP4Scales(__hip_fp8_e4m3_fnuz* blockScalesOut, fpType const* kvVec, SizeType32 size, float globalScale)
{
    constexpr auto quantBlockSize = 16;
    TLLM_CHECK(size % quantBlockSize == 0);

    for (SizeType32 i = 0; i < size / quantBlockSize; ++i)
    {
        auto amax = 0.f;
        for (SizeType32 j = 0; j < quantBlockSize; ++j)
        {
            amax = std::max(amax, std::abs(static_cast<float>(kvVec[i * 16 + j])));
        }
        auto sfValue = globalScale * amax / 6.f;
        blockScalesOut[i] = __hip_fp8_e4m3_fnuz(sfValue);
    }
}

template <typename fpType>
uint8_t quantizeFP4(fpType val, __hip_fp8_e4m3_fnuz blockScale, float globalScale)
{
    float blockScaleFP32 = float(blockScale);

    float outputScale = 0.f;
    if (blockScaleFP32 != 0.f)
    {
        outputScale = 1.f / (blockScaleFP32 / globalScale);
    }

    float quantizedValFP32 = outputScale * static_cast<float>(val);
    auto result = __hip_fp4_e2m1(quantizedValFP32);
    return *reinterpret_cast<uint8_t*>(&result);
}

template <typename fpType>
void quantizeAndCacheFP4(
    __hip_fp4_e2m1* cachePtr, __hip_fp8_e4m3_fnuz* blockScalesPtr, fpType const* kvVec, SizeType32 size, float globalScale)
{
    computeReferenceFP4Scales<fpType>(blockScalesPtr, kvVec, size, globalScale);

    for (SizeType32 i = 0; i < size / 2; ++i)
    {
        constexpr auto quantBlockSize = 16;
        auto blockScale = blockScalesPtr[(2 * i) / quantBlockSize];

        uint8_t val0 = quantizeFP4<fpType>(kvVec[2 * i], blockScale, globalScale);
        uint8_t val1 = quantizeFP4<fpType>(kvVec[2 * i + 1], blockScale, globalScale);

        // Even elements go in low bits, odd go in high
        uint8_t packedVals = (val1 << 4) | val0;
        cachePtr[i] = *reinterpret_cast<__hip_fp4_e2m1*>(&packedVals);
    }
}
#endif

template <typename fpType>
void calculateMeanAndStdDev(fpType const* data, SizeType32 size, float& mean, float& stdDev)
{
    if (size == 0)
    {
        mean = 0.0f;
        stdDev = 0.0f;
        return;
    }

    float sum = 0.0f;
    float sumSq = 0.0f;
    for (SizeType32 i = 0; i < size; ++i)
    {
        float val = static_cast<float>(data[i]);
        sum += val;
        sumSq += val * val;
    }
    mean = sum / size;

    // Variance = E[X^2] - (E[X])^2
    float variance = (sumSq / size) - (mean * mean);
    stdDev = std::sqrt(variance);
}

template <typename fpType>
void calculateMeanAndStdDevOfDifferences(
    fpType const* data1, fpType const* data2, SizeType32 size, float& mean, float& stdDev)
{
    if (size == 0)
    {
        mean = 0.0f;
        stdDev = 0.0f;
        return;
    }

    float sum = 0.0f;
    float sumSq = 0.0f;
    for (SizeType32 i = 0; i < size; ++i)
    {
        float diff = std::abs(static_cast<float>(data1[i]) - static_cast<float>(data2[i]));
        sum += diff;
        sumSq += diff * diff;
    }
    mean = sum / size;

    // Variance = E[X^2] - (E[X])^2
    float variance = (sumSq / size) - (mean * mean);
    stdDev = std::sqrt(variance);
}

inline bool almostEqual(float a, float b, float atol = 1e-2, float rtol = 1e-3)
{
    // Params: a = value to compare and b = reference
    // This function follows implementation of numpy.isclose(), which checks
    //   abs(a - b) <= (atol + rtol * abs(b)).
    // Note that the inequality above is asymmetric where b is considered as
    // a reference value. To account into both absolute/relative errors, it
    // uses absolute tolerance and relative tolerance at the same time. The
    // default values of atol and rtol borrowed from numpy.isclose(). For the
    // case of nan value, the result will be true.
    if (isnan(a) && isnan(b))
    {
        return true;
    }
    return fabs(a - b) <= (atol + rtol * fabs(b));
}

void createCosSinBuf(float* finalValues, SizeType32 numPos, SizeType32 dim, float theta = 10000.0f)
{
    // TODO(dblanaru) fix this function - it generates wrong values
    TLLM_THROW("The createCosSinBuf function contains an error at the moment. Use random initialization instead.");
    // Calculate the inverse frequencies
    std::vector<float> invFreq(dim / 2);
    for (SizeType32 ii = 0; ii < dim / 2; ++ii)
    {
        invFreq[ii] = 1.0f / std::pow(theta, (2.f * ii) / dim);
    }

    // Calculate the sinusoidal inputs and immediately calculate cos and sin values
    SizeType32 index = 0;
    for (SizeType32 pos = 0; pos < numPos; ++pos)
    {
        for (SizeType32 ii = 0; ii < dim / 2; ++ii)
        {
            auto const value = pos * invFreq[ii];
            // Append cos and sin values for each frequency, interleaving them
            finalValues[index++] = std::cos(value);
            finalValues[index++] = std::sin(value);
        }
    }
}

template <typename fpType>
fpType rotateHalfIndex(SizeType32 ii, SizeType32 size, fpType* vec)
{
    auto const halfSize = size / 2;
    auto const sign = (ii < size / 2) ? fpType{-1} : fpType{1};
    return vec[(ii + halfSize) % size] * sign;
}

template <typename fpType>
fpType rotateEveryTwo(SizeType32 ii, SizeType32 size, fpType* vec)
{
    auto sign = (ii % 2) ? fpType{1} : fpType{-1};
    auto offset = (ii % 2) ? SizeType32{-1} : SizeType32{1};
    return vec[ii + offset] * sign;
}

template <typename fpType>
void fillWithOnes(fpType* ptr, SizeType32 sz)
{
    for (SizeType32 ii = 0; ii < sz; ++ii)
    {
        ptr[ii] = static_cast<fpType>(1.0);
    }
}

template <typename fpType>
void fillWithOnesAndZerosInterleaved(fpType* ptr, SizeType32 sz)
{
    for (SizeType32 ii = 0; ii < sz; ii += 2)
    {
        ptr[ii] = static_cast<fpType>(1.0);
        ptr[ii + 1] = static_cast<fpType>(0.0);
    }
}

template <typename fpType>
void applyRopeToBuffer(fpType* srcBuffer, fpType* resBuffer, float2 const* cosSinBuffer, SizeType32 rotaryEmbeddingDim)
{

    for (SizeType32 ii = 0; ii <= rotaryEmbeddingDim; ++ii)
    {
        auto curr = static_cast<float>(srcBuffer[ii]);
        auto currReversed = static_cast<float>(rotateHalfIndex(ii, rotaryEmbeddingDim, srcBuffer));
        auto currFactors = cosSinBuffer[ii % (rotaryEmbeddingDim / 2)];
        auto tmp = currFactors.x * curr + currFactors.y * currReversed;

        resBuffer[ii] = static_cast<float>(tmp);
    }
}

// TODO(dblanaru) add (batch_size, seq_len, 3, head_num, size_per_head) as supported source
// TODO(dblanaru) support different number of kv heads (other than # q heads)
template <typename fpType, typename KVCacheType, typename KVCacheBuffer>
void computeReferenceBiasRope(QKVPreprocessingParams<fpType, KVCacheBuffer> params)
{
    // QKV shape (num_tokens, 3, head_num, size_per_head) in case of non-padded inputs
    // rotary_cos_sin has shape (rotary_embedding_max_positions, rotary_embedding_dim)
    // eg (2048, 128)

    params.setCommonParameters();
    auto const& batchSize = params.batch_size;
    auto const& numTokens = params.token_num;

    // number of query heads
    auto const& qHeadNum = params.head_num;
    // total dimensions of embedding for query heads = qHeadNum*sizePerHead
    auto const& qHiddenSize = params.q_hidden_size;

    // number of key/value heads
    auto const& kvHeadNum = params.kv_head_num;
    // total dimensions of embedding for key/value heads = kvHeadNum*sizePerHead
    auto const& kvHiddenSize = params.kv_hidden_size;

    // dim per head needs to be the same for kv and q for self-attn to work
    auto const& sizePerHead = params.size_per_head;
    // rotaryEmbeddingDim is different for GPT-J and GPT-neox flavors of RoPE
    // for GPT-neox, its just half of the sizePerHead
    // for GPT-J, its an adjustable parameter
    auto const& rotaryEmbeddingDim = params.rotary_embedding_dim;

    // the total size needed for a token (all q+k+v heads)
    auto const& tokenSize = params.hidden_size;

    auto const& kvCache = params.kv_cache_buffer;

    auto tmpResultTensor
        = BufferManager::pinned(ITensor::makeShape({SizeType32(rotaryEmbeddingDim)}), TRTDataType<fpType>::value);
    auto tmpResultPtr = bufferCast<fpType>(*tmpResultTensor);
    // keeps the current token we are looking at (both padded and non-padded version). Needed due to lack of padding
    SizeType32 tokenIt{};

    float kGlobalScale = 0.f;
    float vGlobalScale = 0.f;
    if (params.kv_cache_scale_factors)
    {
        kGlobalScale = params.kv_cache_scale_factors[0];
        vGlobalScale = params.kv_cache_scale_factors[1];
    }

    // the size of a (Q)/K/V matrix TODO(dblanaru) separate this into q and kv sizes

    for (SizeType32 batchIt = 0; batchIt < batchSize; ++batchIt)
    {
        auto const& currContextSize = params.seq_lens[batchIt]; // the context size of the current batch
        for (SizeType32 contextIt = 0; contextIt < currContextSize; ++contextIt)
        {
            // contextIt acts as iterator through the tokens that make up one request

            // currently looking at the beginning of (3, head_num, size_per_head) for a particular token
            // execute this on q (head_num, size_per_head)
            auto const currentCosSinPtr = params.rotary_coef_cache_buffer + contextIt * (rotaryEmbeddingDim / 2);
            for (SizeType32 headIt = 0; headIt < qHeadNum; ++headIt)
            {
                auto const currOffset = tokenIt * tokenSize + headIt * sizePerHead;
                auto const currPtr = params.qkv_input + currOffset;

                applyRopeToBuffer<fpType>(currPtr, tmpResultPtr, currentCosSinPtr, rotaryEmbeddingDim);
                memcpy(currPtr, tmpResultPtr, rotaryEmbeddingDim * sizeof(fpType));
            }

            // do the same for k
            for (SizeType32 headIt = 0; headIt < kvHeadNum; ++headIt)
            {
                auto const currOffset = tokenIt * tokenSize + headIt * sizePerHead + qHiddenSize;
                auto const currPtr = params.qkv_input + currOffset;

                applyRopeToBuffer<fpType>(currPtr, tmpResultPtr, currentCosSinPtr, rotaryEmbeddingDim);
                memcpy(currPtr, tmpResultPtr, rotaryEmbeddingDim * sizeof(fpType));

                auto token_kv_idx = kvCache.getKVTokenIdx(contextIt);
                auto kCachePtr = reinterpret_cast<KVCacheType*>(kvCache.getKBlockPtr(batchIt, token_kv_idx));
                auto offset = kvCache.getKVLocalIdx(contextIt, headIt, sizePerHead, 0);

                if constexpr (std::is_same_v<fpType, KVCacheType>)
                {
                    memcpy(kCachePtr + offset, currPtr, sizePerHead * sizeof(fpType));
                }
                else
                {
#ifdef ENABLE_FP4
                    static_assert(std::is_same_v<KVCacheType, __hip_fp4_e2m1>);

                    auto kBlockScalesCachePtr = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(
                        params.kv_cache_block_scales_buffer.getKBlockPtr(batchIt, token_kv_idx));
                    TLLM_CHECK(sizePerHead % 16 == 0);
                    auto const sizePerHeadBlockScales = sizePerHead / 16;
                    auto const blockScalesOffset = params.kv_cache_block_scales_buffer.getKVLocalIdx(
                        contextIt, headIt, sizePerHeadBlockScales, 0);

                    quantizeAndCacheFP4<fpType>(
                        // We divide the offset by 2 because there are 2 packed FP4
                        // values per byte.
                        reinterpret_cast<__hip_fp4_e2m1*>(kCachePtr + offset / 2),
                        kBlockScalesCachePtr + blockScalesOffset, currPtr, sizePerHead, kGlobalScale);
#else
                    TLLM_THROW("Enable FP4 to run this test.");
#endif
                }
                // dont use tmpResultPtr, but currptr
                // tmpResultPtr will only have {rotaryEmbeddingDim}, but we need {sizePerHead} to also
                // pass the unmodified part of the head to the kv cache
            }

            for (SizeType32 headIt = 0; headIt < kvHeadNum; ++headIt)
            {
                auto const currOffset = tokenIt * tokenSize + headIt * sizePerHead + qHiddenSize + kvHiddenSize;
                auto const currPtr = params.qkv_input + currOffset;

                auto token_kv_idx = kvCache.getKVTokenIdx(contextIt);
                auto vCachePtr = reinterpret_cast<KVCacheType*>(kvCache.getVBlockPtr(batchIt, token_kv_idx));
                auto offset = kvCache.getKVLocalIdx(contextIt, headIt, sizePerHead, 0);
                if constexpr (std::is_same_v<fpType, KVCacheType>)
                {
                    memcpy(vCachePtr + offset, currPtr, sizePerHead * sizeof(fpType));
                }
                else
                {
#ifdef ENABLE_FP4
                    auto vBlockScalesCachePtr = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(
                        params.kv_cache_block_scales_buffer.getVBlockPtr(batchIt, token_kv_idx));
                    TLLM_CHECK(sizePerHead % 16 == 0);
                    auto const sizePerHeadBlockScales = sizePerHead / 16;
                    auto const blockScalesOffset = params.kv_cache_block_scales_buffer.getKVLocalIdx(
                        contextIt, headIt, sizePerHeadBlockScales, 0);

                    quantizeAndCacheFP4<fpType>(reinterpret_cast<__hip_fp4_e2m1*>(vCachePtr + offset / 2),
                        vBlockScalesCachePtr + blockScalesOffset, currPtr, sizePerHead, vGlobalScale);
#else
                    TLLM_THROW("Enable FP4 to run this test.");
#endif
                }
            }
            ++tokenIt;
        }
    }
}

template <typename Pair>
class RopeTest : public testing::Test
{
protected:
    // internal variables
    using FPTypeAndKVCacheType = typename Pair::first_type;
    using KVCacheBuffer = typename Pair::second_type;

    using fpType = typename FPTypeAndKVCacheType::first_type;
    using KVCacheType = typename FPTypeAndKVCacheType::second_type;

    std::shared_ptr<tensorrt_llm::runtime::BufferManager> mBufferManager;
    std::shared_ptr<tensorrt_llm::runtime::CudaStream> mStream;
    BufferManager::ITensorPtr cu_q_seqlens_tensor{nullptr}, cu_kv_seqlens_tensor{nullptr},
        padding_offset_tensor{nullptr}, encoder_padding_offset_tensor{nullptr}, fmha_tile_counter_ptr_tensor{nullptr},
        rotary_inv_freq_buf_tensor{nullptr}, tokens_info_tensor{nullptr}, global_scale_tensor{nullptr};

    std::mt19937 gen;
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // initialize params coming from GPTAttentionPluginCommon
    int mNumHeads{0};
    int mNumKVHeads{0};
    int mHeadSize{0};
    int mRotaryEmbeddingDim{0};
    float mRotaryEmbeddingBase{0.0};
    RotaryScalingType mRotaryEmbeddingScaleType{RotaryScalingType::kNONE};
    float mRotaryEmbeddingScale{0.0};
    int mRotaryEmbeddingMaxPositions{0};
    PositionEmbeddingType mPositionEmbeddingType{PositionEmbeddingType::kROPE_GPT_NEOX};
    bool mRemovePadding{false};
    AttentionMaskType mMaskType{AttentionMaskType::CAUSAL};
    // NOTE: default values for paged kv cache.
    bool mPagedKVCache{false};
    int mTokensPerBlock{0};
    QuantMode mKVCacheQuantMode{};

    bool mCrossAttention{false};

    bool mPosShiftEnabled{false};
    bool mPagedContextFMHA{false};
    bool mFP8ContextFMHA{false};

    // fmha runner (disable by default)
    // flag: disabled = 0, enabled = 1, enabled with fp32 accumulation = 2
    bool mEnableContextFMHA{false};
    int mMultiProcessorCount{0};
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // initialize params coming from runtime - usually packed in EnqueueContextParams
    fpType* attention_input{nullptr};
    fpType const* qkv_bias{nullptr};
    // // Rotary cos sin cache buffer to avoid re-computing.
    BufferManager::ITensorPtr rotary_cos_sin_tensor{nullptr};
    float* rotary_fill_help{nullptr};
    float2 const* rotary_cos_sin{nullptr};

    // TODO(dblanaru) change these to SizeType32
    int32_t input_seq_length{0};
    int32_t max_past_kv_len{0};
    // // By default, max_attention_window == cyclic_attention_window_size
    // // unless each layer has different cyclic kv cache length.
    // // Max cache capacity (used to allocate KV cache)
    int32_t max_attention_window{0};
    // // Cyclic kv cache capacity (used to get the cyclic kv cache position for new tokens)
    int32_t cyclic_attention_window_size{0};
    int32_t sink_token_length{0};
    // these two are actually the same in LLama
    BufferManager::ITensorPtr q_seq_lengths_tensor{nullptr};
    int32_t* q_seq_lengths{nullptr};
    int32_t* kv_seq_lengths{nullptr};
    float* kv_scale_orig_quant{nullptr};
    float* kv_cache_scale_factors{nullptr};

    KVBlockArray::DataType* block_offsets{nullptr};
    void* host_primary_pool_pointer{nullptr};
    void* host_secondary_pool_pointer{nullptr};
    int32_t batch_size{0};
    int32_t num_tokens{0}; // sum of q_seq_lengths
    int32_t max_blocks_per_sequence{0};

    int32_t cross_qkv_length{0};
    int32_t const* encoder_input_lengths{nullptr};

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Other used in `enqueueContext` from `GPTAttentionPluginCommon`
    SizeType32 qkv_size{0};
    QKVPreprocessingParams<fpType, KVCacheBuffer> preprocessingParams;
    float* rotary_inv_freq_buf{nullptr};
    int* tokens_info{nullptr};
    int* cu_q_seqlens{nullptr};
    BufferManager::ITensorPtr attention_input_buf{nullptr};
    KVCacheBuffer keyValueCache, keyValueCacheReference;
    KVCacheBuffer blockScalesCache, blockScalesCacheReference;
    BufferManager::IBufferPtr keyValueCacheBuffer{nullptr}, keyValueCacheBufferReference{nullptr};
    BufferManager::IBufferPtr blockScalesCacheBuffer{nullptr}, blockScalesCacheBufferReference{nullptr};

    void SetUp() override
    {
        if (shouldSkip())
        {
            GTEST_SKIP() << "Skipping due to no/unsupported GPU";
        }

        mStream = std::make_shared<tensorrt_llm::runtime::CudaStream>();
        mBufferManager = std::make_shared<tensorrt_llm::runtime::BufferManager>(mStream);

#ifdef ENABLE_FP4
        if constexpr (std::is_same_v<KVCacheType, __hip_fp4_e2m1>)
        {
            mKVCacheQuantMode += QuantMode::fp4KvCache();
        }
#endif
        gen.seed(42U);
    }

    static bool shouldSkip()
    {
#ifdef ENABLE_FP4
        if constexpr (std::is_same_v<KVCacheType, __hip_fp4_e2m1>)
        {
            // Quant helper functions will not work on lower SM versions.
            return getSMVersion() < 100;
        }
#endif
        return false;
    }

    void allocateBuffers()
    {
        auto const cu_seqlens_size = batch_size + 1;

        cu_q_seqlens_tensor = mBufferManager->pinned(ITensor::makeShape({cu_seqlens_size}), nvinfer1::DataType::kINT32);
        cu_kv_seqlens_tensor
            = mBufferManager->pinned(ITensor::makeShape({cu_seqlens_size}), nvinfer1::DataType::kINT32);
        padding_offset_tensor
            = mBufferManager->pinned(ITensor::makeShape({batch_size, input_seq_length}), nvinfer1::DataType::kINT32);
        encoder_padding_offset_tensor
            = mBufferManager->pinned(ITensor::makeShape({batch_size, cross_qkv_length}), nvinfer1::DataType::kINT32);
        fmha_tile_counter_ptr_tensor
            = mBufferManager->pinned(ITensor::makeShape({mEnableContextFMHA ? 1 : 0}), nvinfer1::DataType::kINT32);
        rotary_inv_freq_buf_tensor = mBufferManager->pinned(
            ITensor::makeShape({batch_size, mRotaryEmbeddingDim / 2}), nvinfer1::DataType::kFLOAT);

        int const max_num_tokens = batch_size * input_seq_length;
        tokens_info_tensor
            = mBufferManager->pinned(ITensor::makeShape({max_num_tokens, 2}), nvinfer1::DataType::kINT32);

#ifdef ENABLE_FP4
        if constexpr (std::is_same_v<KVCacheType, __hip_fp4_e2m1>)
        {
            global_scale_tensor = mBufferManager->pinned(ITensor::makeShape({2}), nvinfer1::DataType::kFLOAT);
        }
#endif
    }

    SizeType32 generateRandomSizeSmallerThan(SizeType32 a)
    {
        // Check if 'a' is less than or equal to 0 to avoid invalid ranges
        if (a <= 0)
        {
            TLLM_CHECK_WITH_INFO(a > 0, "Upped bound of random value must be greater than 0.");
            return 0; // Return an error code or handle as appropriate
        }

        // Define a distribution in the range [0, a-1]
        std::uniform_int_distribution<> distrib(0, a - 1);

        // Generate and return the random number
        return SizeType32{distrib(gen)};
    }

    template <typename fpType>
    void fillRandomNormal(fpType* ptr, SizeType32 sz, float mean = 0.0f, float stdDev = 1.0f)
    {
        std::normal_distribution<float> distr(mean, stdDev);

        for (SizeType32 ii = 0; ii < sz; ++ii)
        {
            ptr[ii] = static_cast<fpType>(distr(gen));
        }
    }

    void setMembersLLama7b()
    {
        mNumHeads = 32;
        mNumKVHeads = 32;
        mHeadSize = 128;
        mRotaryEmbeddingDim = 128;
        mRotaryEmbeddingBase = 10000.0f;
        mRotaryEmbeddingScaleType = RotaryScalingType::kNONE;
        mRotaryEmbeddingScale = 1.0f;
        mRotaryEmbeddingMaxPositions = 2048;
        mPositionEmbeddingType = PositionEmbeddingType::kROPE_GPT_NEOX;
        mRemovePadding = true;
        mMaskType = AttentionMaskType::CAUSAL;

        mPagedKVCache = false;
        mTokensPerBlock = 128;
        mCrossAttention = false;

        mPosShiftEnabled = false;
        mPagedContextFMHA = false;
        mFP8ContextFMHA = false;

        mEnableContextFMHA = true;
        mMultiProcessorCount = tensorrt_llm::common::getMultiProcessorCount();
    }

    void setEnqueueParamsLLama7()
    {
        // // Rotary cos sin cache buffer to avoid re-computing.
        SizeType32 maxOutputSize{generateRandomSizeSmallerThan(1024)};
        rotary_cos_sin_tensor = this->mBufferManager->pinned(
            ITensor::makeShape({mRotaryEmbeddingMaxPositions, mRotaryEmbeddingDim}), nvinfer1::DataType::kFLOAT);
        rotary_fill_help = bufferCast<float>(*(rotary_cos_sin_tensor));
        // createCosSinBuf(rotary_fill_help, mRotaryEmbeddingMaxPositions, mRotaryEmbeddingDim); //currently broken
        // fillWithOnesAndZerosInterleaved(rotary_fill_help, mRotaryEmbeddingMaxPositions*
        // mRotaryEmbeddingDim); //maxes the cos to 1 so it's an identity op
        fillRandomNormal(rotary_fill_help, mRotaryEmbeddingMaxPositions * mRotaryEmbeddingDim);
        rotary_cos_sin = (float2*) (rotary_fill_help);

        batch_size = generateRandomSizeSmallerThan(12);

        q_seq_lengths_tensor = mBufferManager->pinned(ITensor::makeShape({batch_size}), nvinfer1::DataType::kINT32);
        q_seq_lengths = bufferCast<int32_t>(*(q_seq_lengths_tensor));

        for (SizeType32 ii = 0; ii < batch_size; ++ii)
        {
            q_seq_lengths[ii] = generateRandomSizeSmallerThan(1024);
            input_seq_length = std::max(input_seq_length, q_seq_lengths[ii]);
            num_tokens += q_seq_lengths[ii];
        }

        max_past_kv_len = input_seq_length;
        // // By default, max_attention_window == cyclic_attention_window_size
        // // unless each layer has different cyclic kv cache length.
        // // Max cache capacity (used to allocate KV cache)
        max_attention_window = input_seq_length + maxOutputSize;
        // // Cyclic kv cache capacity (used to get the cyclic kv cache position for new tokens)
        cyclic_attention_window_size = input_seq_length + maxOutputSize;
        kv_seq_lengths = q_seq_lengths;

        max_blocks_per_sequence = 0;
        //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    }

    void buildDecoderParams()
    {
        fpType* attention_mask{};
        cu_q_seqlens = bufferCast<int32_t>(*(this->cu_q_seqlens_tensor));
        int* cu_kv_seqlens = bufferCast<int32_t>(*(this->cu_kv_seqlens_tensor));
        int* padding_offset = bufferCast<int32_t>(*(this->padding_offset_tensor));
        int* encoder_padding_offset = bufferCast<int32_t>(*(this->encoder_padding_offset_tensor));
        uint32_t* fmha_tile_counter_ptr = bufferCast<uint32_t>(*(this->fmha_tile_counter_ptr_tensor));
        rotary_inv_freq_buf = bufferCast<float>(*(this->rotary_inv_freq_buf_tensor));
        tokens_info = bufferCast<int>(*(this->tokens_info_tensor));

        BuildDecoderInfoParams<fpType> decoderParams;
        memset(&decoderParams, 0, sizeof(decoderParams));
        decoderParams.seqQOffsets = cu_q_seqlens;
        decoderParams.seqKVOffsets = cu_kv_seqlens;
        decoderParams.paddingOffsets = padding_offset;
        decoderParams.tokensInfo = reinterpret_cast<int2*>(tokens_info);
        decoderParams.encoderPaddingOffsets = mCrossAttention ? encoder_padding_offset : nullptr;
        decoderParams.attentionMask = mCrossAttention ? nullptr : attention_mask; // manually set for cross attn
        // Fixed sequence length offset if not removing the padding (cu_q_seqlens[ii] = ii * seq_length).
        decoderParams.seqQLengths = q_seq_lengths;
        decoderParams.seqKVLengths = mCrossAttention ? encoder_input_lengths : kv_seq_lengths;
        decoderParams.batchSize = batch_size;
        decoderParams.maxQSeqLength = input_seq_length;
        decoderParams.maxEncoderQSeqLength = mCrossAttention ? cross_qkv_length : 0;
        decoderParams.attentionWindowSize = cyclic_attention_window_size;
        decoderParams.sinkTokenLength = sink_token_length;
        decoderParams.numTokens = num_tokens;
        decoderParams.removePadding = mRemovePadding;
        decoderParams.attentionMaskType = mMaskType;
        decoderParams.fmhaTileCounter = fmha_tile_counter_ptr;
        // Rotary embedding inv_freq buffer.
        decoderParams.rotaryEmbeddingScale = mRotaryEmbeddingScale;
        decoderParams.rotaryEmbeddingBase = mRotaryEmbeddingBase;
        decoderParams.rotaryEmbeddingDim = mRotaryEmbeddingDim;
        decoderParams.rotaryScalingType = mRotaryEmbeddingScaleType;
        decoderParams.rotaryEmbeddingInvFreq = rotary_inv_freq_buf;
        decoderParams.rotaryEmbeddingMaxPositions = mRotaryEmbeddingMaxPositions;

        TLLM_LOG_DEBUG(decoderParams.toString());
        invokeBuildDecoderInfo(decoderParams, this->mStream->get());
        sync_check_cuda_error();
    }

    void buildPreprocessingParams()
    {
        bool const enablePagedKVContextFMHA = mPagedKVCache && mPagedContextFMHA;

        KvCacheDataType cache_type = KvCacheDataType::BASE;
        if (mKVCacheQuantMode.hasInt8KvCache())
        {
            cache_type = KvCacheDataType::INT8;
        }
        else if (mKVCacheQuantMode.hasFp8KvCache())
        {
            cache_type = KvCacheDataType::FP8;
        }
        else if (mKVCacheQuantMode.hasFp4KvCache())
        {
            cache_type = KvCacheDataType::NVFP4;
        }

        if (global_scale_tensor)
        {
            kv_cache_scale_factors = bufferCast<float>(*(global_scale_tensor));
            kv_cache_scale_factors[0] = 5.1f;
            kv_cache_scale_factors[1] = 0.25f;
        }

        qkv_size = num_tokens * 3 * mNumHeads * mHeadSize;

        attention_input_buf = this->mBufferManager->pinned(
            ITensor::makeShape({num_tokens, 3, mNumHeads, mHeadSize}), TRTDataType<fpType>::value);
        // ITensor::volume(attention_input_buf->getShape())
        attention_input = bufferCast<fpType>(*attention_input_buf);
        this->fillRandomNormal(attention_input, qkv_size);

        preprocessingParams.qkv_input = const_cast<fpType*>(attention_input);
        preprocessingParams.cross_kv_input = nullptr;
        preprocessingParams.quantized_qkv_output = nullptr; // Assuming this is the correct member for 'O'
        preprocessingParams.q_output = nullptr;
        preprocessingParams.kv_cache_buffer = keyValueCache;
        preprocessingParams.kv_cache_block_scales_buffer = blockScalesCache;
        preprocessingParams.qkv_bias = qkv_bias;
        preprocessingParams.tokens_info = reinterpret_cast<int2*>(tokens_info);
        preprocessingParams.seq_lens = q_seq_lengths;
        preprocessingParams.cache_seq_lens = kv_seq_lengths;
        preprocessingParams.encoder_seq_lens = nullptr;
        preprocessingParams.cu_seq_lens = cu_q_seqlens;
        preprocessingParams.cu_kv_seq_lens = nullptr; // Only used by cross attention.
        preprocessingParams.rotary_embedding_inv_freq = rotary_inv_freq_buf;
        preprocessingParams.rotary_coef_cache_buffer = rotary_cos_sin;
        preprocessingParams.kvScaleOrigQuant = kv_scale_orig_quant;
        preprocessingParams.kv_cache_scale_factors = kv_cache_scale_factors;
        preprocessingParams.spec_decoding_position_offsets = nullptr; // Cast to int* if necessary
        preprocessingParams.batch_size = batch_size;
        preprocessingParams.max_input_seq_len = input_seq_length;
        preprocessingParams.max_kv_seq_len = max_past_kv_len;
        preprocessingParams.cyclic_kv_cache_len = cyclic_attention_window_size;
        preprocessingParams.sink_token_len = sink_token_length;
        preprocessingParams.token_num = num_tokens;
        preprocessingParams.remove_padding = mRemovePadding;
        preprocessingParams.head_num = mNumHeads;
        preprocessingParams.kv_head_num = mNumKVHeads;
        preprocessingParams.qheads_per_kv_head = mNumHeads / mNumKVHeads;
        preprocessingParams.size_per_head = mHeadSize;
        preprocessingParams.rotary_embedding_dim = mRotaryEmbeddingDim;
        preprocessingParams.rotary_embedding_base = mRotaryEmbeddingBase;
        preprocessingParams.rotary_scale_type = mRotaryEmbeddingScaleType;
        preprocessingParams.rotary_embedding_scale = mRotaryEmbeddingScale;
        preprocessingParams.rotary_embedding_max_positions = mRotaryEmbeddingMaxPositions;
        preprocessingParams.position_embedding_type = mPositionEmbeddingType;
        preprocessingParams.position_shift_enabled = mPosShiftEnabled;
        preprocessingParams.cache_type = cache_type;
        preprocessingParams.separate_q_kv_output = enablePagedKVContextFMHA || mCrossAttention;
        preprocessingParams.quantized_fp8_output = mFP8ContextFMHA;
        preprocessingParams.generation_phase = false;
        preprocessingParams.multi_processor_count = mMultiProcessorCount;
        TLLM_CHECK_WITH_INFO(sink_token_length == 0, "sink_token_length != 0 is not supported in the RoPE test.");
    }

    void buildKVCaches()
    {
        size_t elemSizeBits;
        if (mKVCacheQuantMode.hasInt8KvCache() || mKVCacheQuantMode.hasFp8KvCache())
        {
            elemSizeBits = 8;
        }
        else if (mKVCacheQuantMode.hasFp4KvCache())
        {
            elemSizeBits = 4;
        }
        else
        {
            elemSizeBits = sizeof(fpType) * 8;
        }

        auto const sizePerToken = (this->mNumKVHeads * this->mHeadSize * elemSizeBits) / 8;
        auto const totalSize
            = (this->batch_size * 2 * (this->mCrossAttention ? this->cross_qkv_length : this->max_attention_window)
                  * this->mNumKVHeads * this->mHeadSize * elemSizeBits)
            / 8;
        keyValueCacheBuffer = BufferManager::pinned(totalSize);
        void* key_value_cache = static_cast<void*>(keyValueCacheBuffer->data());

        keyValueCacheBufferReference = BufferManager::pinned(totalSize);
        void* key_value_cache_reference = static_cast<void*>(keyValueCacheBufferReference->data());

        void* block_scales_cache = nullptr;
        void* block_scales_cache_reference = nullptr;

        if (mKVCacheQuantMode.hasFp4KvCache())
        {
            TLLM_CHECK_WITH_INFO(
                this->mHeadSize % 16 == 0, "head dim must be divisible by quantization block size (16) for FP4");
            // Dtype of block scales is FP8
            auto const blockScalesTotalSize = this->batch_size * 2
                * (this->mCrossAttention ? this->cross_qkv_length : this->max_attention_window) * this->mNumKVHeads
                * this->mHeadSize / 16;

            blockScalesCacheBuffer = BufferManager::pinned(blockScalesTotalSize);
            block_scales_cache = static_cast<void*>(blockScalesCacheBuffer->data());

            blockScalesCacheBufferReference = BufferManager::pinned(blockScalesTotalSize);
            block_scales_cache_reference = static_cast<void*>(blockScalesCacheBufferReference->data());
        }

        // KVBlockArray::DataType* hostKvCacheBlockOffsets;
        if constexpr (std::is_same_v<KVCacheBuffer, KVBlockArray>)
        {
            TLLM_THROW("Paged KV Cache currently not supported in ropeTest");
        }
        else if constexpr (std::is_same_v<KVCacheBuffer, KVLinearBuffer>)
        {
            // Pointer to the of K/V cache data
            // Shape [B, 2, S*H*D], where 2 is for K and V,
            // B is current number of sequences and
            // H is number of heads
            // S is maximum sequence length
            // D is dimension per head
            // K shape is [B, 1, H, S, D]
            // V shape is [B, 1, H, S, D]
            // NOTE: we have remapped K layout as the same of V.
            using BufferDataType = typename KVCacheBuffer::DataType;
            keyValueCache = KVLinearBuffer(this->batch_size,
                this->mCrossAttention ? this->cross_qkv_length : this->max_attention_window, sizePerToken,
                this->cyclic_attention_window_size, this->sink_token_length, false,
                reinterpret_cast<BufferDataType*>(key_value_cache));

            using BufferDataType = typename KVCacheBuffer::DataType;
            keyValueCacheReference = KVLinearBuffer(this->batch_size,
                this->mCrossAttention ? this->cross_qkv_length : this->max_attention_window, sizePerToken,
                this->cyclic_attention_window_size, this->sink_token_length, false,
                reinterpret_cast<BufferDataType*>(key_value_cache_reference));

            // Block scale buffers are for FP4 only. The shape matches the KV cache pools,
            // except that the block scale is divided by the quantization block size (16
            // for NVFP4).

            if (mKVCacheQuantMode.hasFp4KvCache())
            {
                auto sizePerTokenBlockScales = this->mNumKVHeads * this->mHeadSize / 16;
                blockScalesCache = KVLinearBuffer(this->batch_size,
                    this->mCrossAttention ? this->cross_qkv_length : this->max_attention_window,
                    sizePerTokenBlockScales, this->cyclic_attention_window_size, this->sink_token_length, false,
                    reinterpret_cast<BufferDataType*>(block_scales_cache));

                blockScalesCacheReference = KVLinearBuffer(this->batch_size,
                    this->mCrossAttention ? this->cross_qkv_length : this->max_attention_window,
                    sizePerTokenBlockScales, this->cyclic_attention_window_size, this->sink_token_length, false,
                    reinterpret_cast<BufferDataType*>(block_scales_cache_reference));
            }
        }
    }
};

using RopeTypes = ::testing::Types<
#ifdef ENABLE_FP4
    std::pair<std::pair<half, __hip_fp4_e2m1>, KVLinearBuffer>,
    std::pair<std::pair<hip_bfloat16, __hip_fp4_e2m1>, KVLinearBuffer>,
#endif
    std::pair<std::pair<half, half>, KVLinearBuffer>, std::pair<std::pair<hip_bfloat16, hip_bfloat16>, KVLinearBuffer>,
    std::pair<std::pair<float, float>, KVLinearBuffer>>;

TYPED_TEST_SUITE(RopeTest, RopeTypes);

TYPED_TEST(RopeTest, RopeTestLLamaLinearCache)
{
    using fpType = typename TestFixture::fpType;
    using KVCacheType = typename TestFixture::KVCacheType;
    using KVCacheBuffer = typename TestFixture::KVCacheBuffer;

    this->setMembersLLama7b();
    this->setEnqueueParamsLLama7();

    this->allocateBuffers();

    sync_check_cuda_error();

    this->buildDecoderParams();
    this->buildKVCaches();
    this->buildPreprocessingParams();

    TLLM_LOG_DEBUG(this->preprocessingParams.toString());

    bool allEqual{true};
    BufferManager::ITensorPtr reference_qkv_buf
        = this->mBufferManager->copyFrom(*(this->attention_input_buf), tensorrt_llm::runtime::MemoryType::kPINNEDPOOL);
    fpType* reference_qkv = bufferCast<fpType>(*reference_qkv_buf);

    for (SizeType32 iAssert = 0; iAssert < this->qkv_size; iAssert++)
    {
        if (!almostEqual(static_cast<float>(this->attention_input[iAssert]), static_cast<float>(reference_qkv[iAssert]),
                1e-3, 1e-3))
        {
            TLLM_LOG_ERROR("Mismatch input value. Position of inputs: %d, expected value: %f, output value: %f",
                iAssert, static_cast<float>(this->attention_input[iAssert]),
                static_cast<float>(reference_qkv[iAssert]));
            allEqual = false;
        }
    }
    EXPECT_TRUE(allEqual);

    TLLM_LOG_DEBUG("Parameters generated, random inputs copied. Calling kernel");

    invokeQKVPreprocessing(this->preprocessingParams, this->mStream->get());
    hipDeviceSynchronize();

    this->preprocessingParams.qkv_input = const_cast<fpType*>(reference_qkv);
    this->preprocessingParams.kv_cache_buffer = this->keyValueCacheReference;
    this->preprocessingParams.kv_cache_block_scales_buffer = this->blockScalesCacheReference;
    TLLM_LOG_DEBUG("Kernel finished, calling reference");

    computeReferenceBiasRope<fpType, KVCacheType, KVCacheBuffer>(this->preprocessingParams);
    TLLM_LOG_DEBUG("Reference finished, comparing results");

    hipDeviceSynchronize();
    float mean, stdDev;

    calculateMeanAndStdDev(this->attention_input, this->qkv_size, mean, stdDev);

    TLLM_LOG_DEBUG("Output Mean: %e, Standard Deviation: %e", mean, stdDev);

    calculateMeanAndStdDevOfDifferences(this->attention_input, reference_qkv, this->qkv_size, mean, stdDev);

    TLLM_LOG_DEBUG("Output Abs difference Mean: %e, Standard Deviation: %e", mean, stdDev);
    bool resultsEqual{true};
    for (SizeType32 iAssert = 0; iAssert < this->qkv_size; iAssert++)
    {
        if (!almostEqual(static_cast<float>(this->attention_input[iAssert]), static_cast<float>(reference_qkv[iAssert]),
                1e-5, 0.01f))
        {
            TLLM_LOG_ERROR("Mismatch output value. Position of outputs: %d, expected value: %e, output value: %e",
                iAssert, static_cast<float>(this->attention_input[iAssert]),
                static_cast<float>(reference_qkv[iAssert]));
            resultsEqual = false;
            break;
        }
    }

    EXPECT_TRUE(resultsEqual);

    auto floatKernelKV = reinterpret_cast<KVCacheType*>(this->keyValueCache.data);
    auto floatReferenceKV = reinterpret_cast<KVCacheType*>(this->keyValueCacheReference.data);
    auto const totalSize = this->batch_size * 2
        * (this->mCrossAttention ? this->cross_qkv_length : this->max_attention_window) * this->mNumKVHeads
        * this->mHeadSize;

    auto kernelBlockScales = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(this->blockScalesCache.data);
    auto referenceBlockScales = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(this->blockScalesCacheReference.data);

#ifdef ENABLE_FP4
    constexpr auto isFP4 = std::is_same_v<KVCacheType, __hip_fp4_e2m1>;
#else
    constexpr auto isFP4 = false;
#endif

    auto getKVCacheVal = [&](SizeType32 iAssert, auto* data)
    {
        if constexpr (!isFP4)
        {
            return static_cast<float>(data[iAssert]);
        }
        else
        {
#ifdef ENABLE_FP4
            auto fp4Idx = iAssert / 2;
            auto dataPacked = *reinterpret_cast<uint8_t*>(data + fp4Idx);

            uint8_t storage;
            if (iAssert % 2 == 0)
            {
                storage = dataPacked & 0x0f;
            }
            else
            {
                storage = dataPacked >> 4;
            }
            __hip_fp4_e2m1 val = *reinterpret_cast<__hip_fp4_e2m1*>(&storage);
            return static_cast<float>(val);
#else
            return 0.f;
#endif
        }
    };

    // We allow a very small amount of elements to mismatch for FP4 cases.
    // The computation isn't bit exact compared to the reference, likely due
    // to the fast reciprocal approximation. In our BF16 case, only 2 out of
    // over 100,000,000 elements are mismatched. We leave the tolerance a bit
    // higher than 2 for different random seeds.
    //
    // Note that this approach leads to a more accurate test compared to just
    // increasing the atol to ~0.5 for FP4. With this approach, we make sure that
    // the vast majority of elements match the reference _exactly_ when the dtype
    // is FP4.
    int maxAllowedMismatch = isFP4 ? 10 : 0;
    int numMismatched = 0;
    for (SizeType32 iAssert = 0; iAssert < totalSize; iAssert++)
    {
        auto const ref = getKVCacheVal(iAssert, floatReferenceKV);
        auto const actual = getKVCacheVal(iAssert, floatKernelKV);
        if (!almostEqual(actual, ref, 1e-5, 0.01f))
        {
            TLLM_LOG_ERROR("Mismatch kv cache value. Position in kv cache: %d, expected value: %e, output value: %e",
                iAssert, actual, ref);
            if (++numMismatched > maxAllowedMismatch)
            {
                break;
            }
        }
    }
    EXPECT_TRUE(numMismatched <= maxAllowedMismatch);

    if (isFP4)
    {
        bool kvCacheBlockScalesEqual = true;
        constexpr auto quantBlockSize = 16;
        auto totalSizeBlockScales = totalSize / quantBlockSize;

        for (SizeType32 iAssert = 0; iAssert < totalSizeBlockScales; ++iAssert)
        {
            auto const ref = static_cast<float>(kernelBlockScales[iAssert]);
            auto const actual = static_cast<float>(referenceBlockScales[iAssert]);
            if (!almostEqual(actual, ref, 1e-5, 0.01f))
            {
                TLLM_LOG_ERROR(
                    "Mismatch kv cache block scale value. Position in kv cache: %d, expected value: %e, output value: "
                    "%e",
                    iAssert, actual, ref);
                kvCacheBlockScalesEqual = false;
                break;
            }
        }
        EXPECT_TRUE(kvCacheBlockScalesEqual);
    }
}
