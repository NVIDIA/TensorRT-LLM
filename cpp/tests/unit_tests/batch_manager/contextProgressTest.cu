#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2023-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: NVIDIA TensorRT Source Code License Agreement
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "tensorrt_llm/batch_manager/contextProgress.h"
#include <gmock/gmock.h>
#include <gtest/gtest.h>
#include <thread>

using namespace tensorrt_llm::batch_manager;
using namespace std::chrono;

__global__ void fakeAttention(int* cache, int layerIdx, unsigned computeTimeNs)
{
    // maximum sleep duration: https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#nanosleep-function
    constexpr unsigned maxDuration = 1'000'000;
    while (computeTimeNs > maxDuration)
    {
        __nanosleep(maxDuration);
        computeTimeNs -= maxDuration;
    }
    __nanosleep(computeTimeNs);
    cache[layerIdx] = layerIdx;
}

class ContextProgressTest : public ::testing::Test // NOLINT(cppcoreguidelines-pro-type-member-init)
{
protected:
    void SetUp() override
    {
        hipStreamCreate(&mComputeStream);
        hipStreamCreate(&mExportStream);
    }

    void TearDown() override
    {
        hipStreamDestroy(mComputeStream);
        hipStreamDestroy(mExportStream);
    }

public:
    void runFakePlugin()
    {
        int numLayers = mProgress->getNumLayers();
        for (int i = 0; i < numLayers; i++)
        {
            std::this_thread::sleep_for(mPluginTime);
            fakeAttention<<<1, 1, 0, mComputeStream>>>(mDeviceCache, i, mComputeTime.count());
            EXPECT_EQ(hipGetLastError(), hipSuccess);
            mProgress->recordEvent(i, mComputeStream);
        }
    }

    void receiver()
    {
        int numLayers = mProgress->getNumLayers();
        for (int i = 0; i < numLayers; i++)
        {
            mProgress->wait(i);
            int cache;
            TLLM_CUDA_CHECK(
                hipMemcpyAsync(&cache, mDeviceCache + i, sizeof(int), hipMemcpyDeviceToHost, mExportStream));
            TLLM_CUDA_CHECK(hipStreamSynchronize(mExportStream));
            EXPECT_EQ(cache, i);
            std::this_thread::sleep_for(mTransmissionTime);
        }
    }

    void runFakeModel(int numLayers)
    {
        TLLM_CUDA_CHECK(hipMallocAsync((void**) &mDeviceCache, numLayers * sizeof(int), mComputeStream));
        TLLM_CUDA_CHECK(hipMemsetAsync(mDeviceCache, -1, numLayers * sizeof(int), mComputeStream));
        mProgress = std::make_unique<ContextProgress>(numLayers);

        std::thread receiverThread(&ContextProgressTest::receiver, this);

        runFakePlugin();

        receiverThread.join();
        TLLM_CUDA_CHECK(hipFree(mDeviceCache));
    }

    nanoseconds mPluginTime;
    nanoseconds mComputeTime;
    nanoseconds mTransmissionTime;
    std::unique_ptr<ContextProgress> mProgress;
    hipStream_t mComputeStream;
    hipStream_t mExportStream;
    int* mDeviceCache = nullptr;
};

TEST_F(ContextProgressTest, ContextProgress)
{
    mPluginTime = milliseconds(0);
    mComputeTime = milliseconds(0);
    mTransmissionTime = milliseconds(0);
    runFakeModel(10);
}

TEST_F(ContextProgressTest, SlowPlugin)
{
    mPluginTime = milliseconds(10);
    mComputeTime = milliseconds(1);
    mTransmissionTime = milliseconds(0);
    runFakeModel(10);
}

TEST_F(ContextProgressTest, SlowCompute)
{
    mPluginTime = milliseconds(1);
    mComputeTime = milliseconds(10);
    mTransmissionTime = milliseconds(0);
    runFakeModel(10);
}

TEST_F(ContextProgressTest, SlowTransmission)
{
    mPluginTime = milliseconds(0);
    mComputeTime = milliseconds(1);
    mTransmissionTime = milliseconds(10);
    runFakeModel(10);
}
