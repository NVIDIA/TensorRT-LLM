#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/kernels/quantization.cuh"
#include "userbuffers.h"
#include "utils.h"

namespace tensorrt_llm::kernels::ub
{
using namespace tensorrt_llm::runtime::ub;
#define MAX_THREADS 1024
#define TIMEOUT 200000000000ull

__forceinline__ __device__ int prev_flag(int flag)
{
    return flag > 0 ? (flag - 1) : 2;
}

__forceinline__ __device__ int next_flag(int flag)
{
    return flag < 2 ? (flag + 1) : 0;
}

__forceinline__ __device__ void multi_gpu_block_barrier(int reduce_id, int volatile* flag)
{
#ifdef UB_TIMEOUT_ENABLED
    clock_t s = clock64();
#endif
    while (*flag == prev_flag(reduce_id))
    {
#ifdef UB_TIMEOUT_ENABLED
        if (clock64() - s > 2ull * TIMEOUT)
        {
            printf("NVONLY RSBAR:SM %d [%d]:expecting %d got %d\n", blockIdx.x, threadIdx.x, reduce_id, *flag);
            break;
        }
#endif
    }
}

template <typename DType, int RANKS>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_rw(int const op, int const flagoffset, int const firstrank, int const myrank,
        int const gpustep, size_t const lineoffset, int const numlines, void** commbuff, int const handleridx)
{
#if __CUDA_ARCH__ >= 900
    cudaTriggerProgrammaticLaunchCompletion();
#endif
    __shared__ int4* userptr[RANKS];
    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = next_flag(*reduceidptr);
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
#if __CUDA_ARCH__ >= 900
        cudaGridDependencySynchronize();
#endif
        flagptr[physgpu] = reduce_id;
        userptr[threadIdx.x] = reinterpret_cast<int4*>(commbuff[targetgpu + handleridx]);
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id = next_flag(reduce_id);
    }
    __syncthreads();

    int warp = blockIdx.x + (threadIdx.x >> 5);
    int dest[RANKS];
#pragma unroll
    for (int i = 0; i < RANKS; i++)
        dest[i] = (i + myrank + warp) & (RANKS - 1);

    __syncthreads();
    for (int line = threadIdx.x + blockDim.x * (myrank + RANKS * blockIdx.x); line < numlines;
         line += blockDim.x * gridDim.x * RANKS)
    {
        int4 val[RANKS];

#pragma unroll
        for (int i = 0; i < RANKS; i++)
        {
            val[i] = userptr[dest[i]][lineoffset + line];
        }

        int4 sum = val[0];
        DType* s = reinterpret_cast<DType*>(&sum);

#pragma unroll
        for (int i = 1; i < RANKS; i++)
        {
            DType* x = reinterpret_cast<DType*>(&val[i]);
#pragma unroll
            for (int j = 0; j < 8; j++)
                s[j] += x[j];
        }
#pragma unroll
        for (int i = 0; i < RANKS; i++)
        {
            userptr[dest[i]][lineoffset + line] = sum;
        }
    }

    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // fp16 inplace reduce kernel (Hopper)

template <typename DType, int RANKS>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_rr(int const op, int const flagoffset, int const firstrank, int const myrank,
        int const gpustep, size_t const lineoffset, int const numlines, void** commbuff, int const handleridx)
{
#if __CUDA_ARCH__ >= 900
    cudaTriggerProgrammaticLaunchCompletion();
#endif
    __shared__ int4* userptr[RANKS];
    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = next_flag(*reduceidptr);
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
#if __CUDA_ARCH__ >= 900
        cudaGridDependencySynchronize();
#endif
        flagptr[physgpu] = reduce_id;
        userptr[threadIdx.x] = reinterpret_cast<int4*>(commbuff[targetgpu + handleridx]);
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id = next_flag(reduce_id);
    }
    __syncthreads();

    int warp = blockIdx.x + (threadIdx.x >> 5);
    int dest[RANKS];
#pragma unroll
    for (int i = 0; i < RANKS; i++)
        dest[i] = (i + myrank + warp) & (RANKS - 1);

    __syncthreads();
    for (int line = threadIdx.x + blockDim.x * (myrank + RANKS * blockIdx.x); line < numlines;
         line += blockDim.x * gridDim.x * RANKS)
    {
        int4 val[RANKS];

#pragma unroll
        for (int i = 0; i < RANKS; i++)
        {
            val[i] = userptr[dest[i]][lineoffset + line];
        }

        int4 sum = val[0];
        DType* s = reinterpret_cast<DType*>(&sum);

#pragma unroll
        for (int i = 1; i < RANKS; i++)
        {
            DType* x = reinterpret_cast<DType*>(&val[i]);
#pragma unroll
            for (int j = 0; j < 8; j++)
                s[j] += x[j];
        }

        userptr[myrank][lineoffset + line] = sum;
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }

    int skipmy = 0;
#pragma unroll
    for (int i = 0; i < RANKS; i++)
    {
        int dst = (i + warp + myrank) & (RANKS - 1);
        if (dst == myrank)
        {
            skipmy++;
            continue;
        }
        dest[i - skipmy] = dst;
    }
    __syncthreads();

    for (int line = threadIdx.x + blockDim.x * RANKS * blockIdx.x; line < numlines;
         line += blockDim.x * gridDim.x * RANKS)
    {
        int4 val[RANKS - 1];

#pragma unroll
        for (int i = 0; i < RANKS - 1; i++)
        {
            val[i] = userptr[dest[i]][lineoffset + line + blockDim.x * dest[i]];
        }

#pragma unroll
        for (int i = 0; i < RANKS - 1; i++)
        {
            userptr[myrank][lineoffset + line + blockDim.x * dest[i]] = val[i];
        }
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // fp16 inplace reduce kernel (Ampere)

#if __CUDA_ARCH__ >= 900
template <typename ValType, typename PtrType>
__device__ __forceinline__ void MULTIMEM_ST(ValType val, PtrType ptr)
{
    asm volatile(
        "multimem.st.global.v4.f32 [%0], {%1,%2,%3,%4};" ::"l"(ptr), "r"(val.x), "r"(val.y), "r"(val.z), "r"(val.w)
        : "memory");
}

template <>
__device__ __forceinline__ void MULTIMEM_ST<uint32_t, uint32_t*>(uint32_t val, uint32_t* ptr)
{
    asm volatile("multimem.st.global.b32 [%0], %1;" ::"l"(ptr), "r"(val) : "memory");
}

template <typename ValType, typename PtrType>
__device__ __forceinline__ void MULTIMEM_ST2(ValType& val, PtrType ptr)
{
    asm volatile("multimem.st.global.v2.f32 [%0], {%1,%2};" ::"l"(ptr), "r"(val.x), "r"(val.y) : "memory");
}

template <typename DType, bool const DISABLE_FP32_ACC, typename ValType, typename PtrType>
__device__ __forceinline__ void MULTIMEM_LD(ValType& val, PtrType ptr)
{
    if constexpr (std::is_same_v<DType, half>)
    {
        if (!DISABLE_FP32_ACC)
        {
            asm("multimem.ld_reduce.global.add.v4.f16x2.acc::f32 {%0,%1,%2,%3}, [%4];"
                : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                : "l"(ptr)
                : "memory");
        }
        else
        {
            asm("multimem.ld_reduce.global.add.v4.f16x2 {%0,%1,%2,%3}, [%4];"
                : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                : "l"(ptr)
                : "memory");
        }
    }
#ifdef ENABLE_BF16
    if constexpr (std::is_same_v<DType, __hip_bfloat16>)
    {
        if (!DISABLE_FP32_ACC)
        {
            asm("multimem.ld_reduce.global.add.v4.bf16x2.acc::f32 {%0,%1,%2,%3}, [%4];"
                : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                : "l"(ptr)
                : "memory");
        }
        else
        {
            asm("multimem.ld_reduce.global.add.v4.bf16x2 {%0,%1,%2,%3}, [%4];"
                : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                : "l"(ptr)
                : "memory");
        }
    }
#endif
}

// All MC kernels here
template <typename DType, int RANKS, bool DISABLE_FP32_ACC>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc(int const op, int const flagoffset,
    int const firstrank, int const myrank, int const gpustep, size_t const lineoffset, int const numlines,
    void** commbuff, int const handleridx, float4* mc_ptr)
{
    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = next_flag(*reduceidptr);
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;

        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id = next_flag(reduce_id);
    }
    __syncthreads();
#define UNROLL_MC 8
    int const loop_step0 = blockDim.x * gridDim.x * RANKS;
    int const loop_step = loop_step0 * UNROLL_MC;
    int const start_elem = threadIdx.x + blockDim.x * (myrank + RANKS * blockIdx.x);
    int const end_elem = max(start_elem, numlines);
    int const aligned_elem = ((end_elem - start_elem) / loop_step) * loop_step;
    int const end_aligned = start_elem + aligned_elem;

    for (int line = start_elem; line < end_aligned; line += loop_step)
    {
        uint4 val[UNROLL_MC];
#pragma unroll
        for (int i = 0; i < UNROLL_MC; i++)
            MULTIMEM_LD<DType, DISABLE_FP32_ACC>(val[i], mc_ptr + (lineoffset + line + i * loop_step0));
#pragma unroll
        for (int i = 0; i < UNROLL_MC; i++)
            MULTIMEM_ST(val[i], mc_ptr + (lineoffset + line + i * loop_step0));
    }
    for (int line = end_aligned; line < end_elem; line += loop_step0)
    {
        uint4 val;
        MULTIMEM_LD<DType, DISABLE_FP32_ACC>(val, mc_ptr + (lineoffset + line));
        MULTIMEM_ST(val, mc_ptr + (lineoffset + line));
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // fp16 inplace reduce kernel (Hopper) MC

#else
template <typename DType, int RANKS, bool DISABLE_FP32_ACC>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc(int const op, int const flagoffset,
    int const firstrank, int const myrank, int const gpustep, size_t const lineoffset, int const numlines,
    void** commbuff, int const handleridx, float4* mc_ptr)
{
    printf("userbuffer based kernels not implemented when SM < 90\n");
    asm volatile("brkpt;\n");
}

#endif

#define callranks(x)                                                                                                   \
    if (ar_nvsize == x)                                                                                                \
    {                                                                                                                  \
        int arg1 = userbuffers_allreduceop_nonsharp2 - MAX_OPS, arg2 = REG0_OFFSET(comm) - REG0_SINGLENODE + MAX_OPS,  \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8;                                                                                      \
        int arg7 = elements / 8;                                                                                       \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* kernelArgs[]                                                                                             \
            = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2), reinterpret_cast<void*>(&arg3),         \
                reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5), reinterpret_cast<void*>(&arg6),        \
                reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8), reinterpret_cast<void*>(&arg9)};       \
        TLLM_CUDA_CHECK(hipLaunchKernelExC(&cfg,                                                                      \
            (void*) (comm->use_rr_kernel ? userbuffers_fp16_sum_inplace_gpu_rr<DType, x>                               \
                                         : userbuffers_fp16_sum_inplace_gpu_rw<DType, x>),                             \
            kernelArgs));                                                                                              \
    }

#define callranksMC(x)                                                                                                 \
    if (ar_nvsize == x)                                                                                                \
    {                                                                                                                  \
        int arg1 = userbuffers_allreduceop_nonsharp2 - MAX_OPS, arg2 = REG0_OFFSET(comm) - REG0_SINGLENODE + MAX_OPS,  \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8;                                                                                      \
        int arg7 = elements / 8;                                                                                       \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10)};                                          \
        TLLM_CUDA_CHECK(hipLaunchKernelExC(                                                                           \
            &cfg, (void*) (userbuffers_fp16_sum_inplace_gpu_mc<DType, x, DISABLE_FP32_ACC>), kernelArgs));             \
    }

struct LaunchConfig
{
    LaunchConfig(communicator* comm, int sms, int threads, hipStream_t stream)
    {
        cfg.gridDim = sms;
        cfg.blockDim = threads;
        cfg.dynamicSmemBytes = 0;
        cfg.stream = stream;
        attribute[0].id = hipLaunchAttributeCooperative;
        attribute[1].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attribute[1].val.programmaticStreamSerializationAllowed = comm->pdl_launch;
        attribute[2].id = cudaLaunchAttributeClusterDimension;
        attribute[2].val.clusterDim.x = sms % comm->cga_size == 0 ? comm->cga_size : 1;
        attribute[2].val.clusterDim.y = 1;
        attribute[2].val.clusterDim.z = 1;
        cfg.attrs = attribute;
        cfg.numAttrs = comm->sm_arch >= 9 ? 3 : 1;
    }

    hipLaunchConfig_t& get()
    {
        return cfg;
    }

    hipLaunchConfig_t cfg;
    hipLaunchAttribute attribute[3];
};

template <typename DType>
__inline__ __device__ float compute_rmsnorm2(float val, float s_variance, DType const* gamma, DType const* beta, int i)
{
    float ret = val * s_variance * (float) (gamma[i]);
    if (beta != nullptr)
    {
        ret = ret + (float) (beta[i]);
    }
    return ret;
}

#define SHARD_TOKENS(ntokens, nranks, myrank)                                                                          \
    int first_token = 0, my_tokens;                                                                                    \
    {                                                                                                                  \
        int remapped_rank = myrank;                                                                                    \
        my_tokens = ntokens / nranks;                                                                                  \
        int extra_tokens = ntokens % nranks;                                                                           \
        first_token = remapped_rank * my_tokens;                                                                       \
        first_token += remapped_rank < extra_tokens ? remapped_rank : extra_tokens;                                    \
        if (remapped_rank < extra_tokens)                                                                              \
            my_tokens++;                                                                                               \
    }

// Quantizes the provided PackedVec into the uint32_t output
template <class Type, bool UE8M0_SF = false>
__device__ uint32_t cvt_warp_fp16_to_fp4_mc(PackedVec<Type>& vec, float SFScaleVal, uint8_t* SFout)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
    // Get absolute maximum values among the local 8 values.
    auto localMax = __habs2(vec.elts[0]);

// Local maximum value.
#pragma unroll
    for (int i = 1; i < CVT_FP4_ELTS_PER_THREAD / 2; i++)
    {
        localMax = __hmax2(localMax, __habs2(vec.elts[i]));
    }

    // Get the absolute maximum among all 16 values (two threads).
    localMax = __hmax2(__shfl_xor_sync(uint32_t(-1), localMax, 1), localMax);
    // Get the final absolute maximum values.
    float vecMax = float(__hmax(localMax.x, localMax.y));

    // Get the SF (max value of the vector / max value of e2m1).
    // maximum value of e2m1 = 6.0.
    // TODO: use half as compute data type.
    float SFValue = SFScaleVal * (vecMax * reciprocal_approximate_ftz(6.0f));
    // 8 bits representation of the SF.
    uint8_t fp8SFVal;
    // Write the SF to global memory (STG.8).
    if constexpr (UE8M0_SF)
    {
        // Extract the 8 exponent bits from float32.
        // float 32bits = 1 sign bit + 8 exponent bits + 23 mantissa bits.
        uint32_t tmp = reinterpret_cast<uint32_t&>(SFValue) >> 23;
        fp8SFVal = tmp & 0xff;
        // Convert back to fp32.
        reinterpret_cast<uint32_t&>(SFValue) = tmp << 23;
    }
    else
    {
        // Here SFValue is always positive, so E4M3 is the same as UE4M3.
        __hip_fp8_e4m3_fnuz tmp = __hip_fp8_e4m3_fnuz(SFValue);
        reinterpret_cast<__hip_fp8_e4m3_fnuz&>(fp8SFVal) = tmp;
        // Convert back to fp32.
        SFValue = float(tmp);
    }
    // Get the output scale.
    // Recipe: final_scale = reciprocal(fp32(fp8(SFValue * SFScaleVal))) * reciprocal(SFScaleVal))
    float outputScale
        = SFValue != 0 ? reciprocal_approximate_ftz(SFValue * reciprocal_approximate_ftz(SFScaleVal)) : 0.0f;

    if (threadIdx.x % 2 == 0)
    {
        // Write the SF to global memory (STG.8).
        // *SFout = fp8SFVal;
        uint32_t SFValVec4 = 0;
        uint8_t* SFPtr = reinterpret_cast<uint8_t*>(&SFValVec4);
        SFPtr[(threadIdx.x % 8) / 2] = fp8SFVal;
        SFValVec4 |= __shfl_xor_sync(0x55555555, SFValVec4, 2);
        SFValVec4 |= __shfl_xor_sync(0x55555555, SFValVec4, 4);
        if (threadIdx.x % 8 == 0)
        {
            MULTIMEM_ST(SFValVec4, reinterpret_cast<uint32_t*>(SFout));
        }
    }

    // Convert the input to float.
    float2 fp2Vals[CVT_FP4_ELTS_PER_THREAD / 2];

#pragma unroll
    for (int i = 0; i < CVT_FP4_ELTS_PER_THREAD / 2; i++)
    {
        if constexpr (std::is_same_v<Type, half>)
        {
            fp2Vals[i] = __half22float2(vec.elts[i]);
        }
        else
        {
            fp2Vals[i] = __bfloat1622float2(vec.elts[i]);
        }
        fp2Vals[i].x *= outputScale;
        fp2Vals[i].y *= outputScale;
    }

    // Convert to e2m1 values.
    uint32_t e2m1Vec = fp32_vec_to_e2m1(fp2Vals);

    // Write the e2m1 values to global memory.
    return e2m1Vec;
#else
    return 0;
#endif
}

template <typename DType, int UNROLL_NLINES, bool DISABLE_FP32_ACC>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_fp4(int const op, int const flagoffset, int const firstrank,
        int const myrank, int const gpustep, size_t const lineoffset, int const numlines, void** commbuff,
        int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma, float const eps, int const RANKS,
        uint32_t* mc_ptr_out, size_t const out_lineoffset, float const* scale, uint4* residual_in, uint4* residual_out,
        int res_offset, uint32_t* scale_out, size_t const scale_out_offset, int first_token)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
    using PackedVec = PackedVec<DType>;
    cudaTriggerProgrammaticLaunchCompletion();
    float const sf = 1.f / *scale;
    __shared__ float s_variance;
    int hidden_dim = blockDim.x * UNROLL_NLINES * sizeof(int4) / sizeof(DType);

    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = next_flag(*reduceidptr);
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
        cudaGridDependencySynchronize();
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id = next_flag(reduce_id);
    }
    __syncthreads();

    int const loop_step0 = blockDim.x;
    int const loop_step = loop_step0 * UNROLL_NLINES * gridDim.x;
    int const start_elem = threadIdx.x + blockDim.x * blockIdx.x * UNROLL_NLINES;
    int const end_elem = max(start_elem, numlines);
    int token_idx = first_token + blockIdx.x;
    for (int line = start_elem; line < end_elem; line += loop_step, token_idx += gridDim.x)
    {
        uint4 val[UNROLL_NLINES];
        DType* x = reinterpret_cast<DType*>(&val[0]);
#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            MULTIMEM_LD<DType, DISABLE_FP32_ACC>(val[i], mc_ptr + (lineoffset + line + i * loop_step0));

        if (residual_in != nullptr)
        {
#pragma unroll
            for (int i = 0; i < UNROLL_NLINES; i++)
            {
                uint4 resval = residual_in[res_offset + line + i * loop_step0];
                DType* y = reinterpret_cast<DType*>(&resval);
#pragma unroll
                for (int j = 0; j < 8; j++)
                    x[i * 8 + j] += y[j];
                residual_out[res_offset + line + i * loop_step0] = val[i];
            }
        }

        float local_var_sum = 0.0f;
        for (int j = 0; j < UNROLL_NLINES * sizeof(int4) / sizeof(DType); j++)
            local_var_sum += (float) (x[j]) * (float) (x[j]);

        float packed[1] = {local_var_sum};
        blockReduceSumV2<float, 1>(packed);
        float variance = packed[0];

        if (threadIdx.x == 0)
        {
            variance = (variance / hidden_dim); // Var[x] = E[x²]
            s_variance = rsqrtf(variance + eps);
        }
        __syncthreads();

        int i = 0;
        PackedVec valout;
        DType* y = reinterpret_cast<DType*>(&valout);
#pragma unroll
        for (int g = 0; g < UNROLL_NLINES; g++)
        {
#pragma unroll
            for (int j = 0; j < sizeof(int4) / sizeof(DType); j++)
            {
                y[j] = static_cast<DType>(compute_rmsnorm2<DType>((float) x[i], s_variance, gamma, beta,
                    (threadIdx.x + g * loop_step0) * sizeof(int4) / sizeof(DType) + j));
                i++;
            }
            uint8_t* sf_out = nullptr;
            if (threadIdx.x % 8 == 0)
            {
                sf_out = cvt_quant_to_fp4_get_sf_out_offset<uint32_t, 2>(
                    token_idx, threadIdx.x + g * loop_step0, hidden_dim, scale_out + scale_out_offset);
            }
            uint32_t val = cvt_warp_fp16_to_fp4_mc(valout, sf, sf_out);
            MULTIMEM_ST(val, mc_ptr_out + (out_lineoffset + line + g * loop_step0));
        }
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
#endif
}

template <typename DType, int UNROLL_NLINES, bool DISABLE_FP32_ACC>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_fp4_oneshot(int const op, int const flagoffset,
        int const firstrank, int const myrank, int const gpustep, size_t const lineoffset, int const numlines,
        void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma, float const eps,
        int const RANKS, uint32_t* mc_ptr_out, size_t const out_lineoffset, float const* scale, uint4* residual_in,
        uint4* residual_out, int res_offset, uint32_t* scale_out, size_t const scale_out_offset)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
    using PackedVec = PackedVec<DType>;
    cudaTriggerProgrammaticLaunchCompletion();
    float const sf = 1.f / *scale;
    __shared__ float s_variance;
    int hidden_dim = blockDim.x * UNROLL_NLINES * sizeof(int4) / sizeof(DType);

    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = next_flag(*reduceidptr);
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
        cudaGridDependencySynchronize();
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    __syncthreads();

    int const loop_step0 = blockDim.x;
    int const loop_step = loop_step0 * UNROLL_NLINES * gridDim.x;
    int const start_elem = threadIdx.x + blockDim.x * blockIdx.x * UNROLL_NLINES;
    int const end_elem = max(start_elem, numlines);
    int token_idx = blockIdx.x;
    for (int line = start_elem; line < end_elem; line += loop_step, token_idx += gridDim.x)
    {
        uint4 val[UNROLL_NLINES];
        DType* x = reinterpret_cast<DType*>(&val[0]);
#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            MULTIMEM_LD<DType, DISABLE_FP32_ACC>(val[i], mc_ptr + (lineoffset + line + i * loop_step0));

        if (residual_in != nullptr)
        {
#pragma unroll
            for (int i = 0; i < UNROLL_NLINES; i++)
            {
                uint4 resval = residual_in[res_offset + line + i * loop_step0];
                DType* y = reinterpret_cast<DType*>(&resval);
#pragma unroll
                for (int j = 0; j < 8; j++)
                    x[i * 8 + j] += y[j];
                residual_out[res_offset + line + i * loop_step0] = val[i];
            }
        }

        float local_var_sum = 0.0f;
        for (int j = 0; j < UNROLL_NLINES * sizeof(int4) / sizeof(DType); j++)
            local_var_sum += (float) (x[j]) * (float) (x[j]);

        float packed[1] = {local_var_sum};
        blockReduceSumV2<float, 1>(packed);
        float variance = packed[0];

        if (threadIdx.x == 0)
        {
            variance = (variance / hidden_dim); // Var[x] = E[x²]
            s_variance = rsqrtf(variance + eps);
        }
        __syncthreads();

        int i = 0;
        PackedVec valout;
        DType* y = reinterpret_cast<DType*>(&valout);

#pragma unroll
        for (int g = 0; g < UNROLL_NLINES; g++)
        {
#pragma unroll
            for (int j = 0; j < sizeof(int4) / sizeof(DType); j++)
            {
                y[j] = static_cast<DType>(compute_rmsnorm2<DType>((float) x[i], s_variance, gamma, beta,
                    (threadIdx.x + g * loop_step0) * sizeof(int4) / sizeof(DType) + j));
                i++;
            }
            auto sf_out = cvt_quant_to_fp4_get_sf_out_offset<uint32_t, 2>(
                token_idx, threadIdx.x + g * loop_step0, hidden_dim, scale_out + scale_out_offset);
            mc_ptr_out[out_lineoffset + line + g * loop_step0] = cvt_warp_fp16_to_fp4(valout, sf, sf_out);
        }
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
#endif
}

#if __CUDA_ARCH__ >= 900

template <typename DType, int UNROLL_NLINES, bool DISABLE_FP32_ACC>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant(int const op,
    int const flagoffset, int const firstrank, int const myrank, int const gpustep, size_t const lineoffset,
    int const numlines, void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma,
    float const eps, int const RANKS, float2* mc_ptr_out, size_t const out_lineoffset, float const* scale,
    uint4* residual_in, uint4* residual_out, int res_offset)
{
    cudaTriggerProgrammaticLaunchCompletion();
    float const sf = 1.f / (*scale);
    __shared__ float s_variance;
    int hidden_dim = blockDim.x * UNROLL_NLINES * sizeof(int4) / sizeof(DType);

    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = next_flag(*reduceidptr);
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
        cudaGridDependencySynchronize();
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
        reduce_id = next_flag(reduce_id);
    }
    __syncthreads();

    int const loop_step0 = blockDim.x;
    int const loop_step = loop_step0 * UNROLL_NLINES * gridDim.x;
    int const start_elem = threadIdx.x + blockDim.x * blockIdx.x * UNROLL_NLINES;
    int const end_elem = max(start_elem, numlines);

    for (int line = start_elem; line < end_elem; line += loop_step)
    {
        uint4 val[UNROLL_NLINES];
        DType* x = reinterpret_cast<DType*>(&val[0]);
#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            MULTIMEM_LD<DType, DISABLE_FP32_ACC>(val[i], mc_ptr + (lineoffset + line + i * loop_step0));

        if (residual_in != nullptr)
        {
#pragma unroll
            for (int i = 0; i < UNROLL_NLINES; i++)
            {
                uint4 resval = residual_in[res_offset + line + i * loop_step0];
                DType* y = reinterpret_cast<DType*>(&resval);
#pragma unroll
                for (int j = 0; j < 8; j++)
                    x[i * 8 + j] += y[j];
                residual_out[res_offset + line + i * loop_step0] = val[i];
            }
        }

        float local_var_sum = 0.0f;
        for (int j = 0; j < UNROLL_NLINES * sizeof(int4) / sizeof(DType); j++)
            local_var_sum += (float) (x[j]) * (float) (x[j]);

        float packed[1] = {local_var_sum};
        blockReduceSumV2<float, 1>(packed);
        float variance = packed[0];

        if (threadIdx.x == 0)
        {
            variance = (variance / hidden_dim); // Var[x] = E[x²]
            s_variance = rsqrtf(variance + eps);
        }
        __syncthreads();

        int i = 0;
        uint2 valout;
        __hip_fp8_e4m3_fnuz* y = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(&valout);
#pragma unroll
        for (int g = 0; g < UNROLL_NLINES; g++)
        {
#pragma unroll
            for (int j = 0; j < sizeof(int4) / sizeof(DType); j++)
            {
                y[j] = cuda_cast<__hip_fp8_e4m3_fnuz>(sf
                    * compute_rmsnorm2<DType>((float) x[i], s_variance, gamma, beta,
                        (threadIdx.x + g * loop_step0) * sizeof(int4) / sizeof(DType) + j));
                i++;
            }
            MULTIMEM_ST2(valout, mc_ptr_out + (out_lineoffset + line + g * loop_step0));
        }
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // quant kernel fp16->fp8 twoshot

template <typename DType, int UNROLL_NLINES, bool DISABLE_FP32_ACC>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_oneshot(int const op,
    int const flagoffset, int const firstrank, int const myrank, int const gpustep, size_t const lineoffset,
    int const numlines, void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma,
    float const eps, int const RANKS, uint2* mc_ptr_out, size_t const out_lineoffset, float const* scale,
    uint4* residual_in, uint4* residual_out, int res_offset)
{
    cudaTriggerProgrammaticLaunchCompletion();
    float const sf = 1.f / (*scale);
    __shared__ float s_variance;
    int hidden_dim = blockDim.x * UNROLL_NLINES * sizeof(int4) / sizeof(DType);

    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = next_flag(*reduceidptr);
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
        cudaGridDependencySynchronize();
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    __syncthreads();

    int const loop_step0 = blockDim.x;
    int const loop_step = loop_step0 * UNROLL_NLINES * gridDim.x;
    int const start_elem = threadIdx.x + blockDim.x * blockIdx.x * UNROLL_NLINES;
    int const end_elem = max(start_elem, numlines);

    for (int line = start_elem; line < end_elem; line += loop_step)
    {
        uint4 val[UNROLL_NLINES];
        DType* x = reinterpret_cast<DType*>(&val[0]);
#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            MULTIMEM_LD<DType, DISABLE_FP32_ACC>(val[i], mc_ptr + (lineoffset + line + i * loop_step0));

        if (residual_in != nullptr)
        {
#pragma unroll
            for (int i = 0; i < UNROLL_NLINES; i++)
            {
                uint4 resval = residual_in[res_offset + line + i * loop_step0];
                DType* y = reinterpret_cast<DType*>(&resval);
#pragma unroll
                for (int j = 0; j < 8; j++)
                    x[i * 8 + j] += y[j];
                residual_out[res_offset + line + i * loop_step0] = val[i];
            }
        }

        float local_var_sum = 0.0f;
        for (int j = 0; j < UNROLL_NLINES * sizeof(int4) / sizeof(DType); j++)
            local_var_sum += (float) (x[j]) * (float) (x[j]);

        float packed[1] = {local_var_sum};
        blockReduceSumV2<float, 1>(packed);
        float variance = packed[0];

        if (threadIdx.x == 0)
        {
            variance = (variance / hidden_dim); // Var[x] = E[x²]
            s_variance = rsqrtf(variance + eps);
        }
        __syncthreads();

        int i = 0;
        uint2 valout;
        __hip_fp8_e4m3_fnuz* y = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(&valout);

#pragma unroll
        for (int g = 0; g < UNROLL_NLINES; g++)
        {
#pragma unroll
            for (int j = 0; j < sizeof(int4) / sizeof(DType); j++)
            {
                y[j] = cuda_cast<__hip_fp8_e4m3_fnuz>(sf
                    * compute_rmsnorm2<DType>((float) x[i], s_variance, gamma, beta,
                        (threadIdx.x + g * loop_step0) * sizeof(int4) / sizeof(DType) + j));
                i++;
            }
            mc_ptr_out[out_lineoffset + line + g * loop_step0] = valout;
        }
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // quant kernel fp16->fp8 oneshot

template <typename DType, int UNROLL_NLINES>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_mc_res_allgather(int const op, int const flagoffset, int const firstrank,
        int const myrank, int const gpustep, size_t const lineoffset, int const numlines, void** commbuff,
        int const handleridx, float4* mc_ptr, int const RANKS, uint4* residual_in, int res_offset)
{
    cudaTriggerProgrammaticLaunchCompletion();
    cudaGridDependencySynchronize();
    int *flagptr, physgpu, targetgpu, *myptr;
    int *reduceidptr, reduce_id;
    if (threadIdx.x < RANKS)
    {
        physgpu = myrank * gpustep + firstrank;
        targetgpu = threadIdx.x * gpustep + firstrank;
        int const blockflagoffset = MAX_NVLINK * 2 * blockIdx.x;
        myptr = (reinterpret_cast<int*>(commbuff[physgpu])) + flagoffset;
        reduceidptr = myptr - MAX_OPS;
        reduce_id = next_flag(*reduceidptr);
        flagptr = (reinterpret_cast<int*>(commbuff[targetgpu])) + flagoffset + blockflagoffset;
        myptr += blockflagoffset;
    }
    __syncthreads();

    int const loop_step0 = blockDim.x;
    int const loop_step = loop_step0 * UNROLL_NLINES * gridDim.x;
    int const start_elem = threadIdx.x + blockDim.x * blockIdx.x * UNROLL_NLINES;
    int const end_elem = max(start_elem, numlines);

    for (int line = start_elem; line < end_elem; line += loop_step)
    {
        uint4 val[UNROLL_NLINES];

#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            val[i] = residual_in[res_offset + line + i * loop_step0];

#pragma unroll
        for (int i = 0; i < UNROLL_NLINES; i++)
            MULTIMEM_ST(val[i], mc_ptr + (lineoffset + line + i * loop_step0));
    }
    __syncthreads();
    if (threadIdx.x == 0)
        __threadfence_system();
    __syncthreads();

    if (threadIdx.x < RANKS)
    {
        flagptr[physgpu] = reduce_id;
        multi_gpu_block_barrier(reduce_id, (int volatile*) &myptr[targetgpu]);
    }
    if (threadIdx.x == 0 && blockIdx.x == 0)
        *reduceidptr = reduce_id;
} // residual allgather kernel

#else
template <typename DType, int UNROLL_NLINES, bool DISABLE_FP32_ACC>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant(int const op,
    int const flagoffset, int const firstrank, int const myrank, int const gpustep, size_t const lineoffset,
    int const numlines, void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma,
    float const eps, int const RANKS, float2* mc_ptr_out, size_t const out_lineoffset, float const* scale,
    uint4* residual_in, uint4* residual_out, int res_offset)
{
    printf("userbuffer based kernels not implemented when SM < 90\n");
    asm volatile("brkpt;\n");
}

template <typename DType, int UNROLL_NLINES>
__global__ void __launch_bounds__(MAX_THREADS)
    userbuffers_fp16_sum_inplace_gpu_mc_res_allgather(int const op, int const flagoffset, int const firstrank,
        int const myrank, int const gpustep, size_t const lineoffset, int const numlines, void** commbuff,
        int const handleridx, float4* mc_ptr, int const RANKS, uint4* residual_in, int res_offset)
{
    printf("userbuffer based kernels not implemented when SM < 90\n");
    asm volatile("brkpt;\n");
}

template <typename DType, int UNROLL_NLINES, bool DISABLE_FP32_ACC>
__global__ void __launch_bounds__(MAX_THREADS) userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_oneshot(int const op,
    int const flagoffset, int const firstrank, int const myrank, int const gpustep, size_t const lineoffset,
    int const numlines, void** commbuff, int const handleridx, float4* mc_ptr, DType const* beta, DType const* gamma,
    float const eps, int const RANKS, uint2* mc_ptr_out, size_t const out_lineoffset, float const* scale,
    uint4* residual_in, uint4* residual_out, int res_offset)
{
    printf("userbuffer based kernels not implemented when SM < 90\n");
    asm volatile("brkpt;\n");
}

#endif

#define callranksMC_RMSNORM_QUANT(x)                                                                                   \
    if (nlines == x)                                                                                                   \
    {                                                                                                                  \
        int arg1 = userbuffers_allreduceop_nonsharp2 - MAX_OPS, arg2 = REG0_OFFSET(comm) - REG0_SINGLENODE + MAX_OPS,  \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8 + first_token * hidden_lines;                                                         \
        int arg7 = hidden_lines * my_tokens;                                                                           \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        DType* arg11 = (DType*) beta;                                                                                  \
        DType* arg12 = (DType*) gamma;                                                                                 \
        float arg13 = eps;                                                                                             \
        int arg14 = ar_nvsize;                                                                                         \
        void* arg15 = comm->mc_ptr[out_handler];                                                                       \
        size_t arg16 = out_offset / 8 + first_token * hidden_lines;                                                    \
        float* arg17 = scalefactor;                                                                                    \
        void* arg18 = residual_in;                                                                                     \
        void* arg19 = residual_out;                                                                                    \
        int arg20 = first_token * hidden_lines;                                                                        \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10), reinterpret_cast<void*>(&arg11),          \
            reinterpret_cast<void*>(&arg12), reinterpret_cast<void*>(&arg13), reinterpret_cast<void*>(&arg14),         \
            reinterpret_cast<void*>(&arg15), reinterpret_cast<void*>(&arg16), reinterpret_cast<void*>(&arg17),         \
            reinterpret_cast<void*>(&arg18), reinterpret_cast<void*>(&arg19), reinterpret_cast<void*>(&arg20)};        \
        TLLM_CUDA_CHECK(hipLaunchKernelExC(&cfg,                                                                      \
            (void*) (userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant<DType, x, DISABLE_FP32_ACC>), kernelArgs));     \
    }

#define callranksMC_RMSNORM_QUANT_ONESHOT(x)                                                                           \
    if (nlines == x)                                                                                                   \
    {                                                                                                                  \
        int arg1 = userbuffers_allreduceop_nonsharp2 - MAX_OPS, arg2 = REG0_OFFSET(comm) - REG0_SINGLENODE + MAX_OPS,  \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8;                                                                                      \
        int arg7 = elements / 8;                                                                                       \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        DType* arg11 = (DType*) beta;                                                                                  \
        DType* arg12 = (DType*) gamma;                                                                                 \
        float arg13 = eps;                                                                                             \
        int arg14 = ar_nvsize;                                                                                         \
        void* arg15 = comm->mem_ptr[out_handler];                                                                      \
        size_t arg16 = out_offset / 8;                                                                                 \
        float* arg17 = scalefactor;                                                                                    \
        void* arg18 = residual_in;                                                                                     \
        void* arg19 = residual_out;                                                                                    \
        int arg20 = 0;                                                                                                 \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10), reinterpret_cast<void*>(&arg11),          \
            reinterpret_cast<void*>(&arg12), reinterpret_cast<void*>(&arg13), reinterpret_cast<void*>(&arg14),         \
            reinterpret_cast<void*>(&arg15), reinterpret_cast<void*>(&arg16), reinterpret_cast<void*>(&arg17),         \
            reinterpret_cast<void*>(&arg18), reinterpret_cast<void*>(&arg19), reinterpret_cast<void*>(&arg20)};        \
        TLLM_CUDA_CHECK(hipLaunchKernelExC(&cfg,                                                                      \
            (void*) (userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_oneshot<DType, x, DISABLE_FP32_ACC>),           \
            kernelArgs));                                                                                              \
    }

#define callranksMC_RMSNORM_QUANT_FP4(x)                                                                               \
    if (nlines == x)                                                                                                   \
    {                                                                                                                  \
        int arg1 = userbuffers_allreduceop_nonsharp2 - MAX_OPS, arg2 = REG0_OFFSET(comm) - REG0_SINGLENODE + MAX_OPS,  \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8 + first_token * hidden_lines;                                                         \
        int arg7 = hidden_lines * my_tokens;                                                                           \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        DType* arg11 = (DType*) beta;                                                                                  \
        DType* arg12 = (DType*) gamma;                                                                                 \
        float arg13 = eps;                                                                                             \
        int arg14 = ar_nvsize;                                                                                         \
        void* arg15 = comm->mc_ptr[out_handler];                                                                       \
        size_t arg16 = out_offset / 4 + first_token * hidden_lines;                                                    \
        float* arg17 = scalefactor;                                                                                    \
        void* arg18 = residual_in;                                                                                     \
        void* arg19 = residual_out;                                                                                    \
        int arg20 = first_token * hidden_lines;                                                                        \
        void* arg21 = comm->mc_ptr[scale_handler];                                                                     \
        size_t arg22 = scale_offset / 4;                                                                               \
        int arg23 = first_token;                                                                                       \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10), reinterpret_cast<void*>(&arg11),          \
            reinterpret_cast<void*>(&arg12), reinterpret_cast<void*>(&arg13), reinterpret_cast<void*>(&arg14),         \
            reinterpret_cast<void*>(&arg15), reinterpret_cast<void*>(&arg16), reinterpret_cast<void*>(&arg17),         \
            reinterpret_cast<void*>(&arg18), reinterpret_cast<void*>(&arg19), reinterpret_cast<void*>(&arg20),         \
            reinterpret_cast<void*>(&arg21), reinterpret_cast<void*>(&arg22), reinterpret_cast<void*>(&arg23)};        \
        TLLM_CUDA_CHECK(hipLaunchKernelExC(&cfg,                                                                      \
            (void*) (userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_fp4<DType, x, DISABLE_FP32_ACC>), kernelArgs)); \
    }

#define callranksMC_RMSNORM_QUANT_FP4_ONESHOT(x)                                                                       \
    if (nlines == x)                                                                                                   \
    {                                                                                                                  \
        int arg1 = userbuffers_allreduceop_nonsharp2 - MAX_OPS, arg2 = REG0_OFFSET(comm) - REG0_SINGLENODE + MAX_OPS,  \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8;                                                                                      \
        int arg7 = elements / 8;                                                                                       \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        DType* arg11 = (DType*) beta;                                                                                  \
        DType* arg12 = (DType*) gamma;                                                                                 \
        float arg13 = eps;                                                                                             \
        int arg14 = ar_nvsize;                                                                                         \
        void* arg15 = comm->mem_ptr[out_handler];                                                                      \
        size_t arg16 = out_offset / 4;                                                                                 \
        float* arg17 = scalefactor;                                                                                    \
        void* arg18 = residual_in;                                                                                     \
        void* arg19 = residual_out;                                                                                    \
        int arg20 = 0;                                                                                                 \
        void* arg21 = reinterpret_cast<uint8_t*>(comm->ucbase_ptr[scale_handler])                                      \
            + (ar_firstgpu + ar_nvrank) * comm->mem_size[scale_handler];                                               \
        size_t arg22 = scale_offset / 4;                                                                               \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10), reinterpret_cast<void*>(&arg11),          \
            reinterpret_cast<void*>(&arg12), reinterpret_cast<void*>(&arg13), reinterpret_cast<void*>(&arg14),         \
            reinterpret_cast<void*>(&arg15), reinterpret_cast<void*>(&arg16), reinterpret_cast<void*>(&arg17),         \
            reinterpret_cast<void*>(&arg18), reinterpret_cast<void*>(&arg19), reinterpret_cast<void*>(&arg20),         \
            reinterpret_cast<void*>(&arg21), reinterpret_cast<void*>(&arg22)};                                         \
        TLLM_CUDA_CHECK(hipLaunchKernelExC(&cfg,                                                                      \
            (void*) (userbuffers_fp16_sum_inplace_gpu_mc_rmsnorm_quant_fp4_oneshot<DType, x, DISABLE_FP32_ACC>),       \
            kernelArgs));                                                                                              \
    }
#define callranksMC_RES_AG(x)                                                                                          \
    if (nlines == x)                                                                                                   \
    {                                                                                                                  \
        int arg1 = userbuffers_allreduceop_nonsharp2 - MAX_OPS, arg2 = REG0_OFFSET(comm) - REG0_SINGLENODE + MAX_OPS,  \
            arg3 = ar_firstgpu, arg4 = ar_nvrank, arg5 = ar_step;                                                      \
        size_t arg6 = offset / 8 + first_token * hidden_lines;                                                         \
        int arg7 = hidden_lines * my_tokens;                                                                           \
        void** arg8 = (void**) (comm->gpu_ptrs);                                                                       \
        int arg9 = handler * comm->nvsize;                                                                             \
        void* arg10 = comm->mc_ptr[handler];                                                                           \
        int arg11 = ar_nvsize;                                                                                         \
        uint4* arg12 = (uint4*) residual_in;                                                                           \
        int arg13 = first_token * hidden_lines;                                                                        \
        void* kernelArgs[] = {reinterpret_cast<void*>(&arg1), reinterpret_cast<void*>(&arg2),                          \
            reinterpret_cast<void*>(&arg3), reinterpret_cast<void*>(&arg4), reinterpret_cast<void*>(&arg5),            \
            reinterpret_cast<void*>(&arg6), reinterpret_cast<void*>(&arg7), reinterpret_cast<void*>(&arg8),            \
            reinterpret_cast<void*>(&arg9), reinterpret_cast<void*>(&arg10), reinterpret_cast<void*>(&arg11),          \
            reinterpret_cast<void*>(&arg12), reinterpret_cast<void*>(&arg13)};                                         \
        TLLM_CUDA_CHECK(hipLaunchKernelExC(                                                                           \
            &cfg, (void*) (userbuffers_fp16_sum_inplace_gpu_mc_res_allgather<DType, x>), kernelArgs));                 \
    }

template <typename DType, bool DISABLE_FP32_ACC>
int allreduce2_userbuff_inplace_gpu(int const maxcredit, int const handler, size_t const offset, size_t const elements,
    int const blocksize, communicator* comm, hipStream_t stream)
{
    // schedule GPU kernel only
    // CPU/SHARP part is responsibility of caller
    int const ar_firstgpu = comm->tp_first_rank;
    int const ar_step = 1;
    int const ar_nvsize = comm->tp_size;
    int const ar_nvrank = comm->tp_rank;

    if (elements < 8)
        return 0;
    int sms = ar_nvsize == 1 ? 2 : comm->sms;
    int warps = comm->threads / 32;
    if (warps < ar_nvsize)
        warps = ar_nvsize;
    LaunchConfig launch_config(comm, sms, warps * 32, stream);
    auto& cfg = launch_config.get();
    if (comm->use_mc && (comm->memflags[handler] & UB_MEM_MC_CREATED))
    {
        callranksMC(2) callranksMC(4) callranksMC(8)
#ifdef MNNVL
            callranksMC(16) callranksMC(32)
#endif
    }
    else
    {
        callranks(2) callranks(4) callranks(8)
#ifdef MNNVL
            callranks(16) callranks(32)
#endif
    }

    return sms;
}

template <typename DType, bool DISABLE_FP32_ACC>
void allreduce_nonsharp_inplace(
    int const handler, size_t const offset, size_t const elements, communicator* comm, hipStream_t stream)
{
    if (elements < 64)
        return;
    int blocksize = elements * 2;
    int maxcredit = 0;
    int sms;
    if (DISABLE_FP32_ACC)
    {
        sms = allreduce2_userbuff_inplace_gpu<DType, true>(
            maxcredit, handler, offset, elements, blocksize, comm, stream);
    }
    else
    {
        sms = allreduce2_userbuff_inplace_gpu<DType, false>(
            maxcredit, handler, offset, elements, blocksize, comm, stream);
    }
}

template <typename DType, bool DISABLE_FP32_ACC>
void allreduce2_userbuff_inplace(
    int const handler, size_t const offset, size_t const elements, communicator* comm, hipStream_t stream)
{
    allreduce_nonsharp_inplace<DType, DISABLE_FP32_ACC>(handler, offset, elements, comm, stream);
}

bool use_oneshot_kernel(communicator* comm, size_t elements, int hidden_size)
{
    TLLM_CHECK(elements % hidden_size == 0);
    int token_num = elements / hidden_size;
    if (comm->oneshot == 1 && (elements * comm->tp_size <= 131072))
    {
        return true;
    }
    else if (comm->oneshot == 2 && token_num <= comm->oneshot_force_enable_threshold)
    {
        return true;
    }
    else
    {
        return false;
    }
}

template <typename DType, bool DISABLE_FP32_ACC>
int allreduce2_userbuff_inplace_rmsnorm_quant(int const handler, size_t const offset, int const out_handler,
    size_t const out_offset, size_t const elements, int const hidden_size, void* beta, void* gamma, float eps,
    float* scalefactor, void* residual_in, void* residual_out, communicator* comm, hipStream_t stream)
{
    int const ar_firstgpu = comm->tp_first_rank;
    int const ar_step = 1;
    int const ar_nvsize = comm->tp_size;
    int const ar_nvrank = comm->tp_rank;

    if (elements % hidden_size)
        return 0;
    TLLM_CHECK(hidden_size % 8 == 0);
    int hidden_lines = hidden_size / 8;
    SHARD_TOKENS(elements / hidden_size, ar_nvsize, ar_nvrank);

    int sms = ar_nvsize == 1 ? 2 : comm->sms;
    int nthreads = hidden_size / 8;
    int nlines = 1;
    while (nthreads > 1024)
    {
        nlines++;
        TLLM_CHECK(nlines <= 4);
        if ((hidden_size / 8) % nlines == 0)
            nthreads = ((hidden_size / 8)) / nlines;
    }

    LaunchConfig launch_config(comm, sms, nthreads, stream);
    auto& cfg = launch_config.get();
    if (comm->use_mc && (comm->memflags[handler] & UB_MEM_MC_CREATED))
    {
        if (use_oneshot_kernel(comm, elements, hidden_size))
        {
            callranksMC_RMSNORM_QUANT_ONESHOT(1) callranksMC_RMSNORM_QUANT_ONESHOT(2)
                callranksMC_RMSNORM_QUANT_ONESHOT(3) callranksMC_RMSNORM_QUANT_ONESHOT(4)
        }
        else
        {
            callranksMC_RMSNORM_QUANT(1) callranksMC_RMSNORM_QUANT(2) callranksMC_RMSNORM_QUANT(3)
                callranksMC_RMSNORM_QUANT(4)
        }
    }
    else
    {
        TLLM_CHECK(0);
    }

    return sms;
}

template <typename DType, bool DISABLE_FP32_ACC>
int allreduce2_userbuff_inplace_rmsnorm_quant_fp4(int const handler, size_t const offset, int const out_handler,
    size_t const out_offset, int const scale_handler, size_t const scale_offset, size_t const elements,
    int const hidden_size, void* beta, void* gamma, float eps, float* scalefactor, void* residual_in,
    void* residual_out, communicator* comm, hipStream_t stream)
{
    int const ar_firstgpu = comm->tp_first_rank;
    int const ar_step = 1;
    int const ar_nvsize = comm->tp_size;
    int const ar_nvrank = comm->tp_rank;

    if (elements % hidden_size)
        return 0;
    TLLM_CHECK(hidden_size % 8 == 0);
    int hidden_lines = hidden_size / 8;
    SHARD_TOKENS(elements / hidden_size, ar_nvsize, ar_nvrank);

    int sms = ar_nvsize == 1 ? 2 : comm->sms;
    int nthreads = hidden_size / 8;
    int nlines = 1;
    while (nthreads > 1024)
    {
        nlines++;
        TLLM_CHECK(nlines <= 4);
        if ((hidden_size / 8) % nlines == 0)
            nthreads = ((hidden_size / 8)) / nlines;
    }

    LaunchConfig launch_config(comm, sms, nthreads, stream);
    auto& cfg = launch_config.get();
    if (comm->use_mc && (comm->memflags[handler] & UB_MEM_MC_CREATED))
    {
        if (use_oneshot_kernel(comm, elements, hidden_size))
        {
            callranksMC_RMSNORM_QUANT_FP4_ONESHOT(1) callranksMC_RMSNORM_QUANT_FP4_ONESHOT(2)
                callranksMC_RMSNORM_QUANT_FP4_ONESHOT(3) callranksMC_RMSNORM_QUANT_FP4_ONESHOT(4)
        }
        else
        {
            callranksMC_RMSNORM_QUANT_FP4(1) callranksMC_RMSNORM_QUANT_FP4(2) callranksMC_RMSNORM_QUANT_FP4(3)
                callranksMC_RMSNORM_QUANT_FP4(4)
        }
    }
    else
    {
        TLLM_CHECK(0);
    }

    return sms;
}

template <typename DType>
int allgather2_userbuff_residual(int const handler, size_t const offset, size_t const elements, int const hidden_size,
    void* residual_in, communicator* comm, hipStream_t stream)
{
    // schedule GPU kernel only
    // CPU/SHARP part is not supported yet;
    if (use_oneshot_kernel(comm, elements, hidden_size))
    {
        TLLM_CUDA_CHECK(hipMemcpyAsync(reinterpret_cast<uint8_t*>(comm->mem_ptr[handler]) + (offset * 2), residual_in,
            elements * 2, hipMemcpyDeviceToDevice, stream));
        return 0;
    }
    int const ar_firstgpu = comm->tp_first_rank;
    int const ar_step = 1;
    int const ar_nvsize = comm->tp_size;
    int const ar_nvrank = comm->tp_rank;

    if (elements % hidden_size)
        return 0;
    TLLM_CHECK(hidden_size % 8 == 0);
    int hidden_lines = hidden_size / 8;
    SHARD_TOKENS(elements / hidden_size, ar_nvsize, ar_nvrank);

    int sms = ar_nvsize == 1 ? 2 : comm->sms;
    int nthreads = hidden_size / 8;
    int nlines = 1;
    while (nthreads > 1024)
    {
        nlines++;
        TLLM_CHECK(nlines <= 4);
        if ((hidden_size / 8) % nlines == 0)
            nthreads = ((hidden_size / 8)) / nlines;
    }
    LaunchConfig launch_config(comm, sms, nthreads, stream);
    auto& cfg = launch_config.get();
    if (comm->use_mc && (comm->memflags[handler] & UB_MEM_MC_CREATED))
    {
        callranksMC_RES_AG(1) callranksMC_RES_AG(2) callranksMC_RES_AG(3) callranksMC_RES_AG(4)
    }
    else
    {
        TLLM_CHECK(0);
    }

    return sms;
}

void allreduce2_userbuff_inplace_impl(int const handler, size_t const offset, size_t const elements,
    nvinfer1::DataType dataType, communicator* comm, hipStream_t stream)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kHALF:
    {
        if (kDISABLE_FP32_ACCUMULATION)
        {
            allreduce2_userbuff_inplace<half, true>(handler, offset, elements, comm, stream);
        }
        else
        {
            allreduce2_userbuff_inplace<half, false>(handler, offset, elements, comm, stream);
        }
        break;
    }
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
    {
        if (kDISABLE_FP32_ACCUMULATION)
        {
            allreduce2_userbuff_inplace<__hip_bfloat16, true>(handler, offset, elements, comm, stream);
        }
        else
        {
            allreduce2_userbuff_inplace<__hip_bfloat16, false>(handler, offset, elements, comm, stream);
        }
        break;
    }
#endif
    default: TLLM_THROW("Unsupported dataType for allreduce2_userbuff_inplace_impl");
    }
}

int allgather2_userbuff_residual_impl(int const handler, size_t const offset, size_t const elements,
    int const hidden_size, void* residual, nvinfer1::DataType dataType, communicator* comm, hipStream_t stream)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kHALF:
        return allgather2_userbuff_residual<half>(handler, offset, elements, hidden_size, residual, comm, stream);
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        return allgather2_userbuff_residual<__hip_bfloat16>(
            handler, offset, elements, hidden_size, residual, comm, stream);
        break;
#endif
    default: TLLM_THROW("Unsupported dataType for allgather2_userbuff_residual_impl");
    }
}

int allreduce2_userbuff_inplace_rmsnorm_quant_impl(int const handler, size_t const offset, int const out_handler,
    size_t const out_offset, size_t const elements, int const hidden_size, void* beta, void* gamma, float eps,
    float* scalefactor, void* residual_in, void* residual_out, nvinfer1::DataType dataType, communicator* comm,
    hipStream_t stream)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kHALF:
    {
        if (kDISABLE_FP32_ACCUMULATION)
        {
            return allreduce2_userbuff_inplace_rmsnorm_quant<half, true>(handler, offset, out_handler, out_offset,
                elements, hidden_size, beta, gamma, eps, scalefactor, residual_in, residual_out, comm, stream);
        }
        else
        {
            return allreduce2_userbuff_inplace_rmsnorm_quant<half, false>(handler, offset, out_handler, out_offset,
                elements, hidden_size, beta, gamma, eps, scalefactor, residual_in, residual_out, comm, stream);
        }
        break;
    }
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
    {
        if (kDISABLE_FP32_ACCUMULATION)
        {
            return allreduce2_userbuff_inplace_rmsnorm_quant<__hip_bfloat16, true>(handler, offset, out_handler,
                out_offset, elements, hidden_size, beta, gamma, eps, scalefactor, residual_in, residual_out, comm,
                stream);
        }
        else
        {
            return allreduce2_userbuff_inplace_rmsnorm_quant<__hip_bfloat16, false>(handler, offset, out_handler,
                out_offset, elements, hidden_size, beta, gamma, eps, scalefactor, residual_in, residual_out, comm,
                stream);
        }
        break;
    }
#endif
    default: TLLM_THROW("Unsupported dataType for allreduce2_userbuff_inplace_rmsnorm_quant_impl");
    }
}

int allreduce2_userbuff_inplace_rmsnorm_quant_fp4_impl(int const handler, size_t const offset, int const out_handler,
    size_t const out_offset, int const scale_handler, size_t const scale_offset, size_t const elements,
    int const hidden_size, void* beta, void* gamma, float eps, float* scalefactor, void* residual_in,
    void* residual_out, nvinfer1::DataType dataType, communicator* comm, hipStream_t stream)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kHALF:
        if (kDISABLE_FP32_ACCUMULATION)
        {
            return allreduce2_userbuff_inplace_rmsnorm_quant_fp4<half, true>(handler, offset, out_handler, out_offset,
                scale_handler, scale_offset, elements, hidden_size, beta, gamma, eps, scalefactor, residual_in,
                residual_out, comm, stream);
        }
        else
        {
            return allreduce2_userbuff_inplace_rmsnorm_quant_fp4<half, false>(handler, offset, out_handler, out_offset,
                scale_handler, scale_offset, elements, hidden_size, beta, gamma, eps, scalefactor, residual_in,
                residual_out, comm, stream);
        }
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
    {
        if (kDISABLE_FP32_ACCUMULATION)
        {
            return allreduce2_userbuff_inplace_rmsnorm_quant_fp4<__hip_bfloat16, true>(handler, offset, out_handler,
                out_offset, scale_handler, scale_offset, elements, hidden_size, beta, gamma, eps, scalefactor,
                residual_in, residual_out, comm, stream);
        }
        else
        {
            return allreduce2_userbuff_inplace_rmsnorm_quant_fp4<__hip_bfloat16, false>(handler, offset, out_handler,
                out_offset, scale_handler, scale_offset, elements, hidden_size, beta, gamma, eps, scalefactor,
                residual_in, residual_out, comm, stream);
        }
        break;
    }
#endif
    default: TLLM_THROW("Unsupported dataType for allreduce2_userbuff_inplace_rmsnorm_quant_impl");
    }
}
} // namespace tensorrt_llm::kernels::ub
