/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "DevKernel.h"
#include "RoutingKernel.h"
#include "runner.h"
#include "tensorrt_llm/kernels/trtllmGenKernels/batchedGemm/KernelRunner.h"
#include "tensorrt_llm/kernels/trtllmGenKernels/batchedGemm/trtllmGen_bmm_export/trtllm/gen/DtypeDecl.h"
#include "tensorrt_llm/kernels/trtllmGenKernels/batchedGemm/trtllmGen_bmm_export/trtllm/gen/SfLayoutDecl.h"
#include <iostream>
#include <tensorrt_llm/common/assert.h>

namespace tensorrt_llm
{
namespace kernels
{
namespace trtllmGenFp8BlockScaleMoe
{

namespace tg = trtllm::gen;

namespace Routing
{
namespace
{
inline int32_t computeLog2(int32_t val, std::string const& name = "")
{
    int32_t n = val;
    int32_t out = 0;
    while (n >>= 1)
    {
        ++out;
    }
    TLLM_CHECK_WITH_INFO((1 << out) == val, "Expected %s to be a power of 2, got %d", name.c_str(), val);
    return out;
}
} // namespace

Runner::Runner() {}

void Runner::run(void* routingLogits, void* routingBias, int32_t numTokens, int32_t numExperts, int32_t topK,
    int32_t nGroup, int32_t topkGroup, int32_t localExpertOffset, int32_t localNumExperts, float routedScalingFactor,
    int32_t* routingExpertIndexes, int32_t* expertCountHistogram, int32_t* permutedIdxSize,
    int32_t* expandedIdxToPermutedIdx, int32_t* permutedIdxToExpandedIdx, int32_t* permutedIdxToTokenIdx,
    void* expertWeights, int32_t* numTokensPerExpert, int32_t* ctaIdxXyToBatchIdx, int32_t* ctaIdxXyToMnLimit,
    int32_t* numNonExitingCtas, tg::Dtype dtypeElt, bool useRoutingScalesOnInput, bool useDeepSeekFp8,
    hipStream_t stream)
{
    if (topK == 8)
    {
        // FIXME: hardcoded for now
        int32_t tileN = 8;

        moe::dev::routing::Data routingData;
        routingData.mDtypeElt = dtypeElt; // no-op for now as hidden_state is not input
        routingData.mDtypeExpW = tg::Dtype::Bfloat16;
        routingData.mUsePdl = true;

        // output:
        routingData.mPtrExpertIdx = routingExpertIndexes;
        routingData.mPtrExpertCounts = expertCountHistogram;
        routingData.mPtrPermutedIdxSize = permutedIdxSize;
        routingData.mPtrExpandedIdxToPermutedIdx = expandedIdxToPermutedIdx;
        routingData.mPtrPermutedIdxToExpandedIdx = permutedIdxToExpandedIdx;
        routingData.mPtrPermutedIdxToTokenIdx = permutedIdxToTokenIdx;
        routingData.mPtrNumTokensPerExpert = numTokensPerExpert;
        routingData.mPtrExpertWeights = expertWeights;

        routingData.mPtrCtaIdxXyToBatchIdx = ctaIdxXyToBatchIdx;
        routingData.mPtrCtaIdxXyToMnLimit = ctaIdxXyToMnLimit;
        routingData.mPtrNumNonExitingCtas = numNonExitingCtas;
        routingData.mAllToAllRouteAct = false;

        // input:
        // routingData.mPtrRoutingWeights = args.mRoutingWeights;  // routing weights (don't need if not using gemm)
        routingData.mPtrRoutingBias = routingBias;
        routingData.mPtrScores = reinterpret_cast<float*>(routingLogits);
        // routingData.mPtrIn = args.mInputActs;
        routingData.mNumTokens = numTokens;
        // routingData.mHiddenDim = args.mHiddenDim;
        routingData.mNumExperts = numExperts;
        routingData.mNumExpertGroups = nGroup;
        routingData.mNumLimitedGroups = topkGroup;
        routingData.mTopK = topK;
        routingData.mPaddingLog2 = computeLog2(tileN);
        routingData.mLocalExpertsStartIdx = localExpertOffset;
        routingData.mLocalExpertsStrideLog2 = 0;
        routingData.mNumLocalExperts = localNumExperts;
        routingData.mRouteScale = routedScalingFactor;
        routingData.mUseRoutingSoftmax = false;
        moe::dev::routing::run(routingData, stream);
    }
    else if (topK == 1)
    {
        // FIXME: hardcoded for now
        int32_t tileN = 8;

        moe::dev::routingLlama4::Data routingData;
        // routingData.mDtypeElt = dtypeElt; // no-op for now as hidden_state is not input
        routingData.mDtypeExpW = tg::Dtype::Bfloat16;
        routingData.mUsePdl = true;

        // output:
        routingData.mPtrExpertIdx = routingExpertIndexes;
        routingData.mPtrExpertCounts = expertCountHistogram;
        routingData.mPtrPermutedIdxSize = permutedIdxSize;
        routingData.mPtrExpandedIdxToPermutedIdx = expandedIdxToPermutedIdx;
        // routingData.mPtrPermutedIdxToExpandedIdx = permuted_idx_to_expanded_idx;
        routingData.mPtrPermutedIdxToTokenIdx = permutedIdxToTokenIdx;
        // routingData.mPtrNumTokensPerExpert = num_tokens_per_expert;
        routingData.mPtrExpertWeights = expertWeights;

        routingData.mPtrCtaIdxXyToBatchIdx = ctaIdxXyToBatchIdx;
        routingData.mPtrCtaIdxXyToMnLimit = ctaIdxXyToMnLimit;
        routingData.mPtrNumNonExitingCtas = numNonExitingCtas;
        // routingData.mAllToAllRouteAct = false;

        // input:
        // routingData.mPtrRoutingWeights = args.mRoutingWeights;  // routing weights (don't need if not using gemm)
        // routingData.mPtrRoutingBias = routingBias;
        routingData.mPtrScores = routingLogits;
        // routingData.mPtrIn = args.mInputActs;
        routingData.mNumTokens = numTokens;
        // routingData.mHiddenDim = args.mHiddenDim;
        routingData.mNumExperts = numExperts;
        // routingData.mNumExpertGroups = n_group;
        // routingData.mNumLimitedGroups = topk_group;
        routingData.mTopK = topK;
        routingData.mPaddingLog2 = computeLog2(tileN);
        routingData.mLocalExpertsStartIdx = localExpertOffset;
        routingData.mLocalExpertsStrideLog2 = 0;
        routingData.mNumLocalExperts = localNumExperts;
        // routingData.mRouteScale = routed_scaling_factor;
        // routingData.mUseRoutingSoftmax = false;
        moe::dev::routingLlama4::run(routingData, stream);
    }
    else
    {
        TLLM_CHECK_WITH_INFO(false, "top_k can only be 1 or 8.");
    }
}
} // namespace Routing

namespace PermuteGemm1
{

tensorrt_llm::kernels::TrtllmGenBatchedGemmRunnerOptions getOptions(
    trtllm::gen::Dtype dtypeElt, int32_t tileTokensDim, bool useDeepSeekFp8)
{
    tensorrt_llm::kernels::TrtllmGenBatchedGemmRunnerOptions options = {.eltType = dtypeElt,
        .outputType = dtypeElt,
        .deepSeekFp8 = useDeepSeekFp8,
        .fusedAct = !useDeepSeekFp8,
        .routeAct = true,
        .staticBatch = false,
        .transposeMmaOutput = true,
        .tileSize = tileTokensDim,
        .epilogueTileM = useDeepSeekFp8 ? 64 : 128};
    return options;
}

Runner::Runner(trtllm::gen::Dtype dtypeElt, bool useDeepSeekFp8)
    : mDtypeElt(dtypeElt)
    , mRunner(tensorrt_llm::kernels::TrtllmGenBatchedGemmRunner(getOptions(mDtypeElt, mTileTokensDim, useDeepSeekFp8)))
{
}

void Runner::run(void* hiddenState, void* hiddenStateScale, void* weights, void* weightsScale, void* expertWeights,
    float* outputScalesScalar, float* outputScalesGateScalar, void* output, void* outputScale, int32_t topK,
    int32_t hiddenSize, int32_t intermediateSize, int32_t numExperts, int32_t numTokens, int32_t* permutedIdxToTokenIdx,
    int32_t* ptrNumNonExitingCtas, int32_t* ptrTotalNumPaddedTokens, int32_t* ptrCtaIdxXyToBatchIdx,
    int32_t* ptrCtaIdxXyToMnLimit, void* bmm1Workspace, bool useRoutingScalesOnInput, int device, hipStream_t stream)
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    mRunner.run(numTokens, 2 * intermediateSize, hiddenSize, {}, numTokens, numExperts, maxNumCtasInBatchDim,
        hiddenState, hiddenStateScale, weights, weightsScale, expertWeights, /* perTokensSfB */ nullptr,
        outputScalesScalar, outputScalesGateScalar, output, outputScale, permutedIdxToTokenIdx, ptrTotalNumPaddedTokens,
        ptrCtaIdxXyToBatchIdx, ptrCtaIdxXyToMnLimit, ptrNumNonExitingCtas, bmm1Workspace, stream, device);
}

size_t Runner::getWorkspaceSizeInBytes(
    int32_t topK, int32_t hiddenSize, int32_t intermediateSize, int32_t numExperts, int32_t numTokens)
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    return mRunner.getWorkspaceSizeInBytes(
        numTokens, 2 * intermediateSize, hiddenSize, {}, numTokens, numExperts, maxNumCtasInBatchDim);
}
} // namespace PermuteGemm1

namespace Gemm2
{
tensorrt_llm::kernels::TrtllmGenBatchedGemmRunnerOptions getOptions(
    trtllm::gen::Dtype dtypeElt, trtllm::gen::Dtype dtypeOut, int32_t tileTokensDim, bool useDeepSeekFp8)
{
    tensorrt_llm::kernels::TrtllmGenBatchedGemmRunnerOptions options = {.eltType = dtypeElt,
        .outputType = dtypeOut,
        .deepSeekFp8 = useDeepSeekFp8,
        .fusedAct = false,
        .routeAct = false,
        .staticBatch = false,
        .transposeMmaOutput = true,
        .tileSize = tileTokensDim,
        .epilogueTileM = useDeepSeekFp8 ? 64 : 128};
    return options;
}

Runner::Runner(tg::Dtype dtypeElt, tg::Dtype outputDtype, bool useDeepSeekFp8)
    : mDtypeElt(dtypeElt)
    , mOutputDtype(outputDtype)
    , mRunner(tensorrt_llm::kernels::TrtllmGenBatchedGemmRunner(
          getOptions(mDtypeElt, mOutputDtype, mTileTokensDim, useDeepSeekFp8)))
{
}

void Runner::run(void* permutedHiddenState, void* permutedHiddenStateScale, void* weights, void* weightsScale,
    float* outputScalesScalar, void* output, void* outputScale, int32_t topK, int32_t hiddenSize,
    int32_t intermediateSize, int32_t numExperts, int32_t numTokens, int32_t* ptrNumNonExitingCtas,
    int32_t* ptrTotalNumPaddedTokens, int32_t* ptrCtaIdxXyToBatchIdx, int32_t* ptrCtaIdxXyToMnLimit,
    void* bmm2Workspace, int device, hipStream_t stream)
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    mRunner.run(numTokens, hiddenSize, intermediateSize, {}, numTokens, numExperts, maxNumCtasInBatchDim,
        permutedHiddenState, permutedHiddenStateScale, weights, weightsScale, /* perTokensSfA */ nullptr,
        /* perTokensSfB */ nullptr, outputScalesScalar, /* outputScalesGateScalar */ nullptr, output, outputScale,
        /* permutedIdxToTokenIdx */ nullptr, ptrTotalNumPaddedTokens, ptrCtaIdxXyToBatchIdx, ptrCtaIdxXyToMnLimit,
        ptrNumNonExitingCtas, bmm2Workspace, stream, device);
}

size_t Runner::getWorkspaceSizeInBytes(
    int32_t topK, int32_t hiddenSize, int32_t intermediateSize, int32_t numExperts, int32_t numTokens)
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    return mRunner.getWorkspaceSizeInBytes(
        numTokens, hiddenSize, intermediateSize, {}, numTokens, numExperts, maxNumCtasInBatchDim);
}
} // namespace Gemm2

namespace MoE
{
Runner::Runner(trtllm::gen::Dtype dtypeElt, bool useDeepSeekFp8)
    : mPermuteGemm1(PermuteGemm1::Runner(dtypeElt, useDeepSeekFp8))
    , mGemm2(Gemm2::Runner(dtypeElt, tg::Dtype::Bfloat16, useDeepSeekFp8))
{
}

void Runner::setOpsData(MoERunnerArgs const& args, MoEWorkspace const& workspace,
    moe::dev::convertsf::Data& convertSfData, moe::dev::activation::Data& activationData,
    moe::dev::finalize::Data& finalizeData)
{
    // Setup sf conversion data if needed
    convertSfData.inSfPtr = args.hidden_states_scale;
    convertSfData.outSfPtr = workspace.hidden_states_scale_linear;
    convertSfData.hiddenDimSf = args.hidden_size / 16;
    convertSfData.numTokens = args.num_tokens;
    convertSfData.sfLayoutSrc = tg::SfLayout::R128c4;
    convertSfData.sfLayoutDst = tg::SfLayout::Linear;
    convertSfData.mUsePdl = true;

    // Setup activation data
    activationData.mDtypeElt = args.mDtypeElt;
    activationData.mUsePdl = true;
    activationData.mUseDeepSeekFp8 = true;
    activationData.inPtr = workspace.gemm1_output;
    activationData.outPtr = workspace.activation_output;
    activationData.inDqSfsPtr = workspace.gemm1_output_scale;
    activationData.outDqSfsPtr = workspace.activation_output_scale;
    activationData.innerDim = args.intermediate_size * 2;
    activationData.topK = args.top_k;
    activationData.numTokens = args.num_tokens;
    activationData.expandedIdxToPermutedIdx = workspace.expanded_idx_to_permuted_idx;

    activationData.totalNumPaddedTokens = workspace.total_num_padded_tokens;

    // Setup finalize data
    finalizeData.mDtypeElt = args.mDtypeOut;
    finalizeData.mDtypeExpW = args.mDtypeExpW;
    finalizeData.mUsePdl = true;
    finalizeData.mUseDeepSeekFp8 = false;
    finalizeData.inPtr = workspace.gemm2_output;
    finalizeData.outPtr = args.output;
    finalizeData.inDqSfsPtr = workspace.gemm2_output_scale;
    finalizeData.outDqSfsPtr = args.output_scale;
    if (args.mUseRoutingScalesOnInput)
    {
        finalizeData.expertWeightsPtr = nullptr;
    }
    else
    {
        finalizeData.expertWeightsPtr = workspace.expert_weights;
    }
    finalizeData.expandedIdxToPermutedIdx = workspace.expanded_idx_to_permuted_idx;
    finalizeData.numTokens = args.num_tokens;
    finalizeData.numExperts = args.num_experts;
    finalizeData.topK = args.top_k;
    finalizeData.hiddenDim = args.hidden_size;
    finalizeData.totalNumPaddedTokens = workspace.total_num_padded_tokens;
}

std::tuple<int32_t, int32_t> Runner::getWorkspaceSizeInBytes(MoERunnerArgs const& args)
{
    auto workspace_size_fc1 = static_cast<int32_t>(mPermuteGemm1.getWorkspaceSizeInBytes(
        args.top_k, args.hidden_size, args.intermediate_size, args.local_num_experts, args.num_tokens));
    auto workspace_size_fc2 = static_cast<int32_t>(mGemm2.getWorkspaceSizeInBytes(
        args.top_k, args.hidden_size, args.intermediate_size, args.local_num_experts, args.num_tokens));
    return std::make_tuple(workspace_size_fc1, workspace_size_fc2);
}

void Runner::run(MoERunnerArgs const& args, MoEWorkspace const& workspace, int device, hipStream_t stream)
{
    // Setup all operation data
    moe::dev::activation::Data activationData;
    moe::dev::finalize::Data finalizeData;
    moe::dev::convertsf::Data convertSfData;

    setOpsData(args, workspace, convertSfData, activationData, finalizeData);

    void* hidden_states_scale_linear{args.hidden_states_scale};

    mPermuteGemm1.run(args.hidden_states, hidden_states_scale_linear, args.gemm1_weights, args.gemm1_weights_scale,
        workspace.expert_weights, args.output1_scales_scalar, args.output1_scales_gate_scalar, workspace.gemm1_output,
        workspace.gemm1_output_scale, args.top_k, args.hidden_size, args.intermediate_size, args.local_num_experts,
        args.num_tokens, workspace.permuted_idx_to_token_idx, workspace.num_non_exiting_ctas,
        workspace.total_num_padded_tokens, workspace.cta_idx_xy_to_batch_idx, workspace.cta_idx_xy_to_mn_limit,
        workspace.bmm1_workspace, args.mUseRoutingScalesOnInput, device, stream);

    // We do not fuse activation with FC1 for DeepSeek FP8 due to the weights shuffling constraint.
    void* gemm2_input = workspace.gemm1_output;
    void* gemm2_input_scale = workspace.gemm1_output_scale;
    // We do activation only for DeepSeek FP8, as cubins do not have fused activation.
    if (args.mDtypeElt == tg::Dtype::E4m3 && args.mUseDeepSeekFp8)
    {
        // Run activation
        moe::dev::activation::run(activationData, stream);
        gemm2_input = workspace.activation_output;
        gemm2_input_scale = workspace.activation_output_scale;
    }

    // Run gemm2
    mGemm2.run(gemm2_input, gemm2_input_scale, args.gemm2_weights, args.gemm2_weights_scale, args.output2_scales_scalar,
        workspace.gemm2_output, workspace.gemm2_output_scale, args.top_k, args.hidden_size, args.intermediate_size,
        args.local_num_experts, args.num_tokens, workspace.num_non_exiting_ctas, workspace.total_num_padded_tokens,
        workspace.cta_idx_xy_to_batch_idx, workspace.cta_idx_xy_to_mn_limit, workspace.bmm2_workspace, device, stream);

    // Run finalize
    moe::dev::finalize::run(finalizeData, stream);
}
} // namespace MoE

} // namespace trtllmGenFp8BlockScaleMoe
} // namespace kernels
} // namespace tensorrt_llm
