/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "DevKernel.h"
#include "RoutingKernel.h"
#include "runner.h"
#include "tensorrt_llm/kernels/trtllmGenKernels/batchedGemm/KernelRunner.h"
#include "tensorrt_llm/kernels/trtllmGenKernels/batchedGemm/trtllmGen_bmm_export/trtllm/gen/DtypeDecl.h"
#include "tensorrt_llm/kernels/trtllmGenKernels/batchedGemm/trtllmGen_bmm_export/trtllm/gen/SfLayoutDecl.h"
#include <iostream>
#include <tensorrt_llm/common/assert.h>

namespace tensorrt_llm
{
namespace kernels
{
namespace trtllmGenFp8BlockScaleMoe
{

namespace btg = batchedGemm::trtllm::gen;

namespace Routing
{
namespace
{
inline int32_t computeLog2(int32_t val, std::string const& name = "")
{
    int32_t n = val;
    int32_t out = 0;
    while (n >>= 1)
    {
        ++out;
    }
    TLLM_CHECK_WITH_INFO((1 << out) == val, "Expected %s to be a power of 2, got %d", name.c_str(), val);
    return out;
}
} // namespace

Runner::Runner() {}

Runner::Runner(int32_t tileTokensDim)
    : mTileTokensDim(tileTokensDim)
{
}

void Runner::run(void* routingLogits, void* routingBias, int32_t numTokens, int32_t numExperts, int32_t topK,
    int32_t nGroup, int32_t topkGroup, int32_t localExpertOffset, int32_t localNumExperts, float routedScalingFactor,
    int32_t* routingExpertIndexes, int32_t* expertCountHistogram, int32_t* permutedIdxSize,
    int32_t* expandedIdxToPermutedIdx, int32_t* permutedIdxToExpandedIdx, int32_t* permutedIdxToTokenIdx,
    void* expertWeights, int32_t* numTokensPerExpert, int32_t* ctaIdxXyToBatchIdx, int32_t* ctaIdxXyToMnLimit,
    int32_t* numNonExitingCtas, btg::Dtype dtypeElt, bool useRoutingScalesOnInput, bool useDeepSeekFp8,
    RoutingMethodType routingMethodType, hipStream_t stream)
{
    if (routingMethodType == RoutingMethodType::DeepSeekV3)
    {
        TLLM_CHECK_WITH_INFO(topK <= 8, "For DeepSeek routing method, must have topK <= 8");
        TLLM_CHECK_WITH_INFO(topkGroup <= 4, "For DeepSeek routing method, must have topkGroup <= 4");
        moe::dev::routing::Data routingData;
        routingData.mDtypeExpW = btg::Dtype::Bfloat16;
        routingData.mUsePdl = true;

        // output:
        routingData.mPtrExpertIdx = routingExpertIndexes;
        routingData.mPtrExpertCounts = expertCountHistogram;
        routingData.mPtrPermutedIdxSize = permutedIdxSize;
        routingData.mPtrExpandedIdxToPermutedIdx = expandedIdxToPermutedIdx;
        routingData.mPtrPermutedIdxToExpandedIdx = permutedIdxToExpandedIdx;
        routingData.mPtrPermutedIdxToTokenIdx = permutedIdxToTokenIdx;
        routingData.mPtrNumTokensPerExpert = numTokensPerExpert;
        routingData.mPtrExpertWeights = expertWeights;

        routingData.mPtrCtaIdxXyToBatchIdx = ctaIdxXyToBatchIdx;
        routingData.mPtrCtaIdxXyToMnLimit = ctaIdxXyToMnLimit;
        routingData.mPtrNumNonExitingCtas = numNonExitingCtas;
        routingData.mAllToAllRouteAct = false;

        // input:
        // routingData.mPtrRoutingWeights = args.mRoutingWeights;  // routing weights (don't need if not using gemm)
        routingData.mPtrRoutingBias = routingBias;
        routingData.mPtrScores = reinterpret_cast<float*>(routingLogits);
        // routingData.mPtrIn = args.mInputActs;
        routingData.mNumTokens = numTokens;
        // routingData.mHiddenDim = args.mHiddenDim;
        routingData.mNumExperts = numExperts;
        routingData.mNumExpertGroups = nGroup;
        routingData.mNumLimitedGroups = topkGroup;
        routingData.mTopK = topK;
        routingData.mPaddingLog2 = computeLog2(mTileTokensDim);
        routingData.mLocalExpertsStartIdx = localExpertOffset;
        routingData.mLocalExpertsStrideLog2 = 0;
        routingData.mNumLocalExperts = localNumExperts;
        routingData.mRouteScale = routedScalingFactor;
        routingData.mUseRoutingSoftmax = false;
        moe::dev::routing::run(routingData, stream);
    }
    else if (routingMethodType == RoutingMethodType::Llama4)
    {
        TLLM_CHECK_WITH_INFO(topK == 1, "For Llama routing method, must have topK == 1");
        if (nGroup > 0 || topkGroup > 0)
        {
            TLLM_LOG_WARNING("For Llama routing method, nGroup/topkGroup is ignored, got %d/%d.", nGroup, topkGroup);
        }
        moe::dev::routingLlama4::Data routingData;
        routingData.mDtypeExpW = btg::Dtype::Bfloat16;
        routingData.mUsePdl = true;

        // output:
        routingData.mPtrExpertIdx = routingExpertIndexes;
        routingData.mPtrExpertCounts = expertCountHistogram;
        routingData.mPtrPermutedIdxSize = permutedIdxSize;
        routingData.mPtrExpandedIdxToPermutedIdx = expandedIdxToPermutedIdx;
        // routingData.mPtrPermutedIdxToExpandedIdx = permuted_idx_to_expanded_idx;
        routingData.mPtrPermutedIdxToTokenIdx = permutedIdxToTokenIdx;
        // routingData.mPtrNumTokensPerExpert = num_tokens_per_expert;
        routingData.mPtrExpertWeights = expertWeights;

        routingData.mPtrCtaIdxXyToBatchIdx = ctaIdxXyToBatchIdx;
        routingData.mPtrCtaIdxXyToMnLimit = ctaIdxXyToMnLimit;
        routingData.mPtrNumNonExitingCtas = numNonExitingCtas;
        // routingData.mAllToAllRouteAct = false;

        // input:
        // routingData.mPtrRoutingWeights = args.mRoutingWeights;  // routing weights (don't need if not using gemm)
        // routingData.mPtrRoutingBias = routingBias;
        routingData.mPtrScores = routingLogits;
        // routingData.mPtrIn = args.mInputActs;
        routingData.mNumTokens = numTokens;
        // routingData.mHiddenDim = args.mHiddenDim;
        routingData.mNumExperts = numExperts;
        // routingData.mNumExpertGroups = nGroup;
        // routingData.mNumLimitedGroups =topkGroup;
        routingData.mTopK = topK;
        routingData.mPaddingLog2 = computeLog2(mTileTokensDim);
        routingData.mLocalExpertsStartIdx = localExpertOffset;
        routingData.mLocalExpertsStrideLog2 = 0;
        routingData.mNumLocalExperts = localNumExperts;
        // routingData.mRouteScale = routed_scaling_factor;
        // routingData.mUseRoutingSoftmax = false;
        moe::dev::routingLlama4::run(routingData, stream);
    }
    else if (routingMethodType == RoutingMethodType::Renormalize /* default */
        || routingMethodType == RoutingMethodType::RenormalizeNaive /* Softmax -> TopK */)
    {
        moe::dev::routingQwen3::Data routingData;

        //
        // Config
        //

        routingData.mDtypeExpW = btg::Dtype::Bfloat16;
        // routingData.mDtypeElt = dtypeElt; // no-op for now as hidden_state is not input
        routingData.mUsePdl = true;
        routingData.mDoSoftmaxBeforeTopK = routingMethodType == RoutingMethodType::RenormalizeNaive;
        routingData.mNormTopkProb = routingMethodType == RoutingMethodType::RenormalizeNaive;

        routingData.mPtrScores = routingLogits;

        //
        // Outputs
        //
        routingData.mPtrExpertIdx = routingExpertIndexes;
        routingData.mPtrExpertCounts = expertCountHistogram;
        routingData.mPtrPermutedIdxSize = permutedIdxSize;
        routingData.mPtrExpandedIdxToPermutedIdx = expandedIdxToPermutedIdx;
        routingData.mPtrPermutedIdxToTokenIdx = permutedIdxToTokenIdx;
        routingData.mPtrExpertWeights = expertWeights;

        //
        // Grouped Gemm Launch Config Buffers
        //
        routingData.mPtrCtaIdxXyToBatchIdx = ctaIdxXyToBatchIdx;
        routingData.mPtrCtaIdxXyToMnLimit = ctaIdxXyToMnLimit;
        routingData.mPtrNumNonExitingCtas = numNonExitingCtas;

        //
        // Inputs
        //
        routingData.mNumTokens = numTokens;
        routingData.mNumExperts = numExperts;
        routingData.mTopK = topK;
        routingData.mPaddingLog2 = computeLog2(mTileTokensDim);
        routingData.mLocalExpertsStartIdx = localExpertOffset;
        routingData.mLocalExpertsStrideLog2 = 0;
        routingData.mNumLocalExperts = localNumExperts;

        moe::dev::routingQwen3::run(routingData, stream);
    }
    else
    {
        TLLM_CHECK_WITH_INFO(false, "Unimplemented routing method %s of enum %d",
            serializeMoeRoutingMethodType(routingMethodType).c_str(), (int) routingMethodType);
    }
}
} // namespace Routing

namespace PermuteGemm1
{

tensorrt_llm::kernels::TrtllmGenBatchedGemmRunnerOptions getOptions(
    btg::Dtype dtypeElt, int32_t tileTokensDim, bool useDeepSeekFp8)
{
    tensorrt_llm::kernels::TrtllmGenBatchedGemmRunnerOptions options = {.eltType = dtypeElt,
        .outputType = dtypeElt,
        .deepSeekFp8 = useDeepSeekFp8,
        .fusedAct = !useDeepSeekFp8,
        .routeAct = true,
        .staticBatch = false,
        .transposeMmaOutput = true,
        .tileSize = tileTokensDim,
        .epilogueTileM = useDeepSeekFp8 ? 64 : 128};
    return options;
}

Runner::Runner(btg::Dtype dtypeElt, bool useDeepSeekFp8, int tileTokensDim)
    : mDtypeElt(dtypeElt)
    , mTileTokensDim(tileTokensDim)
    , mRunner(tensorrt_llm::kernels::TrtllmGenBatchedGemmRunner(getOptions(mDtypeElt, mTileTokensDim, useDeepSeekFp8)))
{
}

void Runner::run(void* hiddenState, void* hiddenStateScale, void* weights, void* weightsScale, void* expertWeights,
    float* outputScalesScalar, float* outputScalesGateScalar, void* output, void* outputScale, int32_t topK,
    int32_t hiddenSize, int32_t intermediateSize, int32_t numExperts, int32_t numTokens, int32_t* permutedIdxToTokenIdx,
    int32_t* ptrNumNonExitingCtas, int32_t* ptrTotalNumPaddedTokens, int32_t* ptrCtaIdxXyToBatchIdx,
    int32_t* ptrCtaIdxXyToMnLimit, void* bmm1Workspace, bool useRoutingScalesOnInput, int device, hipStream_t stream,
    int32_t configIndex)
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    mRunner.run(numTokens, 2 * intermediateSize, hiddenSize, {}, numTokens, numExperts, maxNumCtasInBatchDim,
        hiddenState, hiddenStateScale, weights, weightsScale, expertWeights, /* perTokensSfB */ nullptr,
        outputScalesScalar, outputScalesGateScalar, output, outputScale, permutedIdxToTokenIdx, ptrTotalNumPaddedTokens,
        ptrCtaIdxXyToBatchIdx, ptrCtaIdxXyToMnLimit, ptrNumNonExitingCtas, bmm1Workspace, stream, device, configIndex);
}

size_t Runner::getWorkspaceSizeInBytes(int32_t topK, int32_t hiddenSize, int32_t intermediateSize, int32_t numExperts,
    int32_t numTokens, int32_t configIndex) const
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    return mRunner.getWorkspaceSizeInBytes(
        numTokens, 2 * intermediateSize, hiddenSize, {}, numTokens, numExperts, maxNumCtasInBatchDim, configIndex);
}

int32_t Runner::getDefaultValidConfigIndex(
    int32_t topK, int32_t hiddenSize, int32_t intermediateSize, int32_t numExperts, int32_t numTokens) const
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    return mRunner.getDefaultValidConfigIndex(
        numTokens, 2 * intermediateSize, hiddenSize, {}, numTokens, numExperts, maxNumCtasInBatchDim);
}

bool Runner::isValidConfigIndex(int32_t configIndex, int32_t topK, int32_t hiddenSize, int32_t intermediateSize,
    int32_t numExperts, int32_t numTokens) const
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);

    auto const isValid = mRunner.isValidConfigIndex(
        configIndex, numTokens, 2 * intermediateSize, hiddenSize, {}, numTokens, numExperts, maxNumCtasInBatchDim);

    return isValid;
}

std::vector<int64_t> Runner::getPassingConfigIndices() const
{
    return mRunner.getPassingConfigIndices();
}

} // namespace PermuteGemm1

namespace Gemm2
{
tensorrt_llm::kernels::TrtllmGenBatchedGemmRunnerOptions getOptions(
    btg::Dtype dtypeElt, btg::Dtype dtypeOut, int32_t tileTokensDim, bool useDeepSeekFp8)
{
    tensorrt_llm::kernels::TrtllmGenBatchedGemmRunnerOptions options = {.eltType = dtypeElt,
        .outputType = dtypeOut,
        .deepSeekFp8 = useDeepSeekFp8,
        .fusedAct = false,
        .routeAct = false,
        .staticBatch = false,
        .transposeMmaOutput = true,
        .tileSize = tileTokensDim,
        .epilogueTileM = useDeepSeekFp8 ? 64 : 128};
    return options;
}

Runner::Runner(btg::Dtype dtypeElt, btg::Dtype outputDtype, bool useDeepSeekFp8, int tileTokensDim)
    : mDtypeElt(dtypeElt)
    , mOutputDtype(outputDtype)
    , mTileTokensDim(tileTokensDim)
    , mRunner(tensorrt_llm::kernels::TrtllmGenBatchedGemmRunner(
          getOptions(mDtypeElt, mOutputDtype, mTileTokensDim, useDeepSeekFp8)))
{
}

void Runner::run(void* permutedHiddenState, void* permutedHiddenStateScale, void* weights, void* weightsScale,
    float* outputScalesScalar, void* output, void* outputScale, int32_t topK, int32_t hiddenSize,
    int32_t intermediateSize, int32_t numExperts, int32_t numTokens, int32_t* ptrNumNonExitingCtas,
    int32_t* ptrTotalNumPaddedTokens, int32_t* ptrCtaIdxXyToBatchIdx, int32_t* ptrCtaIdxXyToMnLimit,
    void* bmm2Workspace, int device, hipStream_t stream, int32_t configIndex)
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    mRunner.run(numTokens, hiddenSize, intermediateSize, {}, numTokens, numExperts, maxNumCtasInBatchDim,
        permutedHiddenState, permutedHiddenStateScale, weights, weightsScale, /* perTokensSfA */ nullptr,
        /* perTokensSfB */ nullptr, outputScalesScalar, /* outputScalesGateScalar */ nullptr, output, outputScale,
        /* permutedIdxToTokenIdx */ nullptr, ptrTotalNumPaddedTokens, ptrCtaIdxXyToBatchIdx, ptrCtaIdxXyToMnLimit,
        ptrNumNonExitingCtas, bmm2Workspace, stream, device, configIndex);
}

size_t Runner::getWorkspaceSizeInBytes(int32_t topK, int32_t hiddenSize, int32_t intermediateSize, int32_t numExperts,
    int32_t numTokens, int32_t configIndex) const
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    return mRunner.getWorkspaceSizeInBytes(
        numTokens, hiddenSize, intermediateSize, {}, numTokens, numExperts, maxNumCtasInBatchDim, configIndex);
}

int32_t Runner::getDefaultValidConfigIndex(
    int32_t topK, int32_t hiddenSize, int32_t intermediateSize, int32_t numExperts, int32_t numTokens) const
{
    auto maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);
    return mRunner.getDefaultValidConfigIndex(
        numTokens, hiddenSize, intermediateSize, {}, numTokens, numExperts, maxNumCtasInBatchDim);
}

bool Runner::isValidConfigIndex(int32_t configIndex, int32_t topK, int32_t hiddenSize, int32_t intermediateSize,
    int32_t numExperts, int32_t numTokens) const
{

    auto const maxNumCtasInBatchDim = Routing::getMaxNumCtasInBatchDim(numTokens, topK, numExperts, mTileTokensDim);

    auto const isValid = mRunner.isValidConfigIndex(
        configIndex, numTokens, hiddenSize, intermediateSize, {}, numTokens, numExperts, maxNumCtasInBatchDim);

    return isValid;
}

std::vector<int64_t> Runner::getPassingConfigIndices() const
{
    return mRunner.getPassingConfigIndices();
}

} // namespace Gemm2

namespace MoE
{
Runner::Runner(btg::Dtype dtypeElt, bool useDeepSeekFp8, int32_t tileTokensDim)
    : mPermuteGemm1(PermuteGemm1::Runner(dtypeElt, useDeepSeekFp8, tileTokensDim))
    , mGemm2(Gemm2::Runner(dtypeElt, btg::Dtype::Bfloat16, useDeepSeekFp8, tileTokensDim))
{

    auto const& gemm1PassingIndices = mPermuteGemm1.getPassingConfigIndices();
    auto const& gemm2PassingIndices = mGemm2.getPassingConfigIndices();

    auto const totalPassingIndices = gemm1PassingIndices.size() * gemm2PassingIndices.size();
    mPassingConfigs.reserve(totalPassingIndices);

    for (auto const& indexGemm1 : gemm1PassingIndices)
    {
        for (auto const& indexGemm2 : gemm2PassingIndices)
        {
            mPassingConfigs.push_back(MoEConfig{indexGemm1, indexGemm2});
        }
    }

    TLLM_CHECK_WITH_INFO(!mPassingConfigs.empty(), "No compatible configs found for the fp8 block scale MoE runner.");
}

void Runner::setOpsData(MoERunnerArgs const& args, MoEWorkspace const& workspace,
    moe::dev::convertsf::Data& convertSfData, moe::dev::activation::Data& activationData,
    moe::dev::finalize::Data& finalizeData)
{
    // Setup sf conversion data if needed
    convertSfData.inSfPtr = args.hidden_states_scale;
    convertSfData.outSfPtr = workspace.hidden_states_scale_linear;
    convertSfData.hiddenDimSf = args.hidden_size / 16;
    convertSfData.numTokens = args.num_tokens;
    convertSfData.sfLayoutSrc = btg::SfLayout::R128c4;
    convertSfData.sfLayoutDst = btg::SfLayout::Linear;
    convertSfData.mUsePdl = true;

    // Setup activation data
    activationData.mDtypeElt = args.mDtypeElt;
    activationData.mUsePdl = true;
    activationData.mUseDeepSeekFp8 = true;
    activationData.inPtr = workspace.gemm1_output;
    activationData.outPtr = workspace.activation_output;
    activationData.inDqSfsPtr = workspace.gemm1_output_scale;
    activationData.outDqSfsPtr = workspace.activation_output_scale;
    activationData.innerDim = args.intermediate_size * 2;
    activationData.topK = args.top_k;
    activationData.numTokens = args.num_tokens;
    activationData.expandedIdxToPermutedIdx = workspace.expanded_idx_to_permuted_idx;

    activationData.totalNumPaddedTokens = workspace.total_num_padded_tokens;

    if (args.do_finalize)
    {
        // Setup finalize data
        finalizeData.mDtypeElt = args.mDtypeOut;
        finalizeData.mDtypeExpW = args.mDtypeExpW;
        finalizeData.mUsePdl = true;
        finalizeData.mUseDeepSeekFp8 = false;
        finalizeData.inPtr = workspace.gemm2_output;
        finalizeData.outPtr = args.output;
        finalizeData.inDqSfsPtr = workspace.gemm2_output_scale;
        finalizeData.outDqSfsPtr = args.output_scale;
        if (args.mUseRoutingScalesOnInput)
        {
            finalizeData.expertWeightsPtr = nullptr;
        }
        else
        {
            finalizeData.expertWeightsPtr = workspace.expert_weights;
        }
        finalizeData.expandedIdxToPermutedIdx = workspace.expanded_idx_to_permuted_idx;
        finalizeData.numTokens = args.num_tokens;
        finalizeData.numExperts = args.num_experts;
        finalizeData.topK = args.top_k;
        finalizeData.hiddenDim = args.hidden_size;
        finalizeData.totalNumPaddedTokens = workspace.total_num_padded_tokens;
    }
}

std::tuple<int32_t, int32_t> Runner::getWorkspaceSizeInBytes(MoERunnerArgs const& args, int64_t configIndex) const
{
    auto const& config = mPassingConfigs[configIndex];

    auto workspace_size_fc1 = static_cast<int32_t>(mPermuteGemm1.getWorkspaceSizeInBytes(args.top_k, args.hidden_size,
        args.intermediate_size, args.local_num_experts, args.num_tokens, config.gemm1Config));
    auto workspace_size_fc2 = static_cast<int32_t>(mGemm2.getWorkspaceSizeInBytes(args.top_k, args.hidden_size,
        args.intermediate_size, args.local_num_experts, args.num_tokens, config.gemm2Config));
    return std::make_tuple(workspace_size_fc1, workspace_size_fc2);
}

std::vector<int64_t> Runner::getValidConfigIndices(
    int32_t topK, int32_t hiddenSize, int32_t intermediateSize, int32_t numLocalExperts, int32_t numTokens) const
{
    std::vector<int64_t> validIndices;

    for (int i = 0; i < mPassingConfigs.size(); ++i)
    {
        auto const& config = mPassingConfigs[i];

        if (mPermuteGemm1.isValidConfigIndex(
                config.gemm1Config, topK, hiddenSize, intermediateSize, numLocalExperts, numTokens)
            && mGemm2.isValidConfigIndex(
                config.gemm2Config, topK, hiddenSize, intermediateSize, numLocalExperts, numTokens))
        {
            validIndices.push_back(i);
        }
    }

    return validIndices;
}

int64_t Runner::getDefaultValidConfigIndex(
    int32_t topK, int32_t hiddenSize, int32_t intermediateSize, int32_t numLocalExperts, int32_t numTokens) const
{
    auto const validIndices = getValidConfigIndices(topK, hiddenSize, intermediateSize, numLocalExperts, numTokens);

    return validIndices[0];
}

void Runner::run(
    MoERunnerArgs const& args, MoEWorkspace const& workspace, int device, hipStream_t stream, int64_t configIndex)
{
    // Setup all operation data
    moe::dev::activation::Data activationData;
    moe::dev::finalize::Data finalizeData;
    moe::dev::convertsf::Data convertSfData;
    sync_check_cuda_error(stream);
    setOpsData(args, workspace, convertSfData, activationData, finalizeData);

    void* hidden_states_scale_linear{args.hidden_states_scale};

    auto const& config = mPassingConfigs[configIndex];

    mPermuteGemm1.run(args.hidden_states, hidden_states_scale_linear, args.gemm1_weights, args.gemm1_weights_scale,
        workspace.expert_weights, args.output1_scales_scalar, args.output1_scales_gate_scalar, workspace.gemm1_output,
        workspace.gemm1_output_scale, args.top_k, args.hidden_size, args.intermediate_size, args.local_num_experts,
        args.num_tokens, workspace.permuted_idx_to_token_idx, workspace.num_non_exiting_ctas,
        workspace.total_num_padded_tokens, workspace.cta_idx_xy_to_batch_idx, workspace.cta_idx_xy_to_mn_limit,
        workspace.bmm1_workspace, args.mUseRoutingScalesOnInput, device, stream, config.gemm1Config);

    // We do not fuse activation with FC1 for DeepSeek FP8 due to the weights shuffling constraint.
    void* gemm2_input = workspace.gemm1_output;
    void* gemm2_input_scale = workspace.gemm1_output_scale;
    // We do activation only for DeepSeek FP8, as cubins do not have fused activation.
    if (args.mDtypeElt == btg::Dtype::E4m3 && args.mUseDeepSeekFp8)
    {
        // Run activation
        moe::dev::activation::run(activationData, stream);
        gemm2_input = workspace.activation_output;
        gemm2_input_scale = workspace.activation_output_scale;
    }

    // Run gemm2
    mGemm2.run(gemm2_input, gemm2_input_scale, args.gemm2_weights, args.gemm2_weights_scale, args.output2_scales_scalar,
        workspace.gemm2_output, workspace.gemm2_output_scale, args.top_k, args.hidden_size, args.intermediate_size,
        args.local_num_experts, args.num_tokens, workspace.num_non_exiting_ctas, workspace.total_num_padded_tokens,
        workspace.cta_idx_xy_to_batch_idx, workspace.cta_idx_xy_to_mn_limit, workspace.bmm2_workspace, device, stream,
        config.gemm2Config);

    // Run finalize
    if (args.do_finalize)
    {
        // Run finalize
        moe::dev::finalize::run(finalizeData, stream);
        sync_check_cuda_error(stream);
    }
}
} // namespace MoE

} // namespace trtllmGenFp8BlockScaleMoe
} // namespace kernels
} // namespace tensorrt_llm
