#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "DevKernel.h"

//// FIX
#include "Utils.h"  // #include <trtllm/dev/Utils.h>
#include "macros.h" // #include <utils/macros.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
// #include <trtllm/gen/GenCtx.h>

#include <hipcub/hipcub.hpp>

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace moe::dev
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace activation
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace tg = trtllm::gen;

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void activationKernel(KernelParams params)
{
    using Type = typename KernelParams::Type;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // immediately trigger the secondary kernel when using PDL, then wait on primary
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
        cudaGridDependencySynchronize();
    }
#endif

    for (int tokenIdx = blockIdx.z; tokenIdx < params.numTokens; tokenIdx += gridDim.z)
    {
        // Look over experts per token
        for (int k = blockIdx.y; k < params.topK; k += gridDim.y)
        {
            int const expandedIdx = tokenIdx * params.topK + k;
            int const permutedIdx = params.expandedIdxToPermutedIdx[expandedIdx];
            if (permutedIdx == -1)
                continue;

            // Loop over hidden dim
            for (int hiddenIdx = threadIdx.x + blockDim.x * blockIdx.x; hiddenIdx < params.innerDim / 2;
                 hiddenIdx += blockDim.x * gridDim.x)
            {
                int const baseIdx = permutedIdx * params.innerDim + hiddenIdx;

                float x1 = (float) params.inPtr[baseIdx];
                float x2 = (float) params.inPtr[baseIdx + params.innerDim / 2];

                float act = trtllm::dev::silu(x2);
                Type out = (Type) (act * x1);

                int const outIdx = permutedIdx * (params.innerDim / 2) + hiddenIdx;
                params.outPtr[outIdx] = out;
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void activationDeepSeekKernel(KernelParams params)
{
    using Type = typename KernelParams::Type;
    using BlockReduce = hipcub::BlockReduce<float, 128>;

    __shared__ float s_scaleOut;
    __shared__ typename BlockReduce::TempStorage temp_storage;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // immediately trigger the secondary kernel when using PDL, then wait on primary
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
        cudaGridDependencySynchronize();
    }
#endif
    // Loop over tokens
    for (int tokenIdx = blockIdx.z; tokenIdx < params.numTokens; tokenIdx += gridDim.z)
    {
        // Look over experts per token
        for (int k = blockIdx.y; k < params.topK; k += gridDim.y)
        {
            int const expandedIdx = tokenIdx * params.topK + k;
            int const permutedIdx = params.expandedIdxToPermutedIdx[expandedIdx];

            // Needed for expert parallelism
            if (permutedIdx == -1)
                continue;

            // Loop over hidden dim
            for (int hiddenIdx = threadIdx.x + blockDim.x * blockIdx.x; hiddenIdx < params.innerDim / 2;
                 hiddenIdx += blockDim.x * gridDim.x)
            {
                int const baseIdx = permutedIdx * params.innerDim + hiddenIdx;

                int const totalNumPaddedTokens = params.totalNumPaddedTokens[0];

                int const scale1_idx = permutedIdx + totalNumPaddedTokens * (hiddenIdx / 128);
                int const scale2_idx
                    = permutedIdx + totalNumPaddedTokens * ((hiddenIdx / 128) + (params.innerDim / 2 / 128));
                float const scale1 = params.inDqSfsPtr[scale1_idx];
                float const scale2 = params.inDqSfsPtr[scale2_idx];

                float x1 = scale1 * (float) params.inPtr[baseIdx];
                float x2 = scale2 * (float) params.inPtr[baseIdx + params.innerDim / 2];

                float act = trtllm::dev::silu(x2);
                float out = act * x1;

                // The largest (finite) value that can be represented using E4m3.
                float constexpr E4m3MaxVal{448.f};

                // Compute the absolute max
                float aMax = BlockReduce(temp_storage).Reduce(fabsf(out), hipcub::Max());
                if (threadIdx.x == 0)
                {
                    s_scaleOut = aMax / E4m3MaxVal;
                    int const scaleOut_idx = permutedIdx + totalNumPaddedTokens * (hiddenIdx / 128);
                    params.outDqSfsPtr[scaleOut_idx] = aMax / E4m3MaxVal;
                }
                __syncthreads();
                float const scaleOut = s_scaleOut;
                __syncthreads();
                int const outIdx = permutedIdx * (params.innerDim / 2) + hiddenIdx;
                params.outPtr[outIdx] = (Type) (out / scaleOut);
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    if (data.mDtypeElt == tg::Dtype::E2m1)
    {
        // Note: this should be unreachable because the options are checked beforehand.
        // E2m1 requires using higher-precision intermediate data (bf16).
        TLLM_LOG_ERROR("Activation with E2m1_t isn't supported.");
        return;
    }

    if (data.mUseDeepSeekFp8)
    {
        int const numThreads = 128;
        const dim3 grid(data.innerDim / 128, data.topK, data.numTokens);

        LAUNCH(data, activationDeepSeekKernel, grid, numThreads, 0, stream);
    }
    else
    {
        int const numThreads = 256;
        const dim3 grid(data.innerDim / 128, data.topK, data.numTokens);

        LAUNCH(data, activationKernel, grid, numThreads, 0, stream);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace activation

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace convertsf
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace tg = trtllm::gen;

namespace dev
{
// Compute the offset that corresponds to (dataRowIdx, dataBlkColIdx) in the SF tensor where
// dataRowIdx and dataBlkColIdx are the respective indices of the row and the block of 16 elts
// from the K dim in the tensor of data.
inline __device__ int64_t getSfOffset(int32_t dataRowIdx, int32_t dataBlkColIdx, int32_t numDataBlksPerRow)
{

    // The number of rows of SF per block.
    static int32_t constexpr NumRowsPerSfBlock = 128;
    // The number of cols of SF per block.
    static int32_t constexpr NumColsPerSfBlock = 4;
    // The size of each SF block.
    static int32_t constexpr NumBytesPerSfBlock = NumRowsPerSfBlock * NumColsPerSfBlock;

    // The number of rows of data per SF block.
    static int32_t constexpr NumDataRowsPerSfBlock = NumRowsPerSfBlock;
    // The number of cols of blocks of data per SF block.
    static int32_t constexpr NumDataBlkColsPerSfBlock = NumColsPerSfBlock;

    // The row of the SF block in the SF tensor.
    int sfBlkRowIdx = dataRowIdx / NumDataRowsPerSfBlock;
    // The col of the SF block in the SF tensor.
    int sfBlkColIdx = dataBlkColIdx / NumDataBlkColsPerSfBlock;
    // The blocks are stored row-major in the tensor of scaling factors.
    int sfBlkIdx = sfBlkRowIdx * numDataBlksPerRow / NumDataBlkColsPerSfBlock + sfBlkColIdx;

    // Find the row in the SF block.
    int sfRowIdx = (dataRowIdx % 32) * 4 + (dataRowIdx % NumDataRowsPerSfBlock) / 32;
    // Find the col in the SF block.
    int sfColIdx = (dataBlkColIdx % 4);

    // Compute the offset in bytes.
    return sfBlkIdx * NumBytesPerSfBlock + sfRowIdx * NumColsPerSfBlock + sfColIdx;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Given the GMEM address of an output element, compute the offset of the corresponding scaling
// factor in the SF tensor. Optionally, a startTokenIndex can be provided if the first token is not
// the start token in the SF tensor. This is useful when inflight batching is enabled in TRT-LLM,
// where the context and generation output are stored as one output tensor. In this case, the
// generation output may not start with zero offset in the SF output tensor.
template <int32_t NumBitsPerElt>
inline __device__ int64_t getSfOffset(int64_t gmemOffsetInBytes, int32_t hiddenDim, int32_t startTokenIdx = 0)
{
    // The number of elements per sf.
    int32_t constexpr NumEltsPerSf = 16;
    // The GMEM offset of the output element.
    int64_t gmemOffset = gmemOffsetInBytes * 8 /*bits*/ / NumBitsPerElt;
    // The row/col indices of the corresponding SF element.
    int32_t sfRowIdx = gmemOffset / hiddenDim + startTokenIdx;
    int32_t sfColIdx = (gmemOffset % hiddenDim) / NumEltsPerSf;
    // Compute the SF offset.
    return getSfOffset(sfRowIdx, sfColIdx, hiddenDim / NumEltsPerSf);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// TODO(tizheng): Refactor to track gmem offset instead of doing pointer subtraction.
template <int32_t NumBitsPerElt>
inline __device__ int64_t getSfOffset(
    void const* gmemOutPtr, void const* gmemBasePtr, int32_t hiddenDim, int32_t startTokenIdx = 0)
{
    return getSfOffset<NumBitsPerElt>(
        reinterpret_cast<char const*>(gmemOutPtr) - reinterpret_cast<char const*>(gmemBasePtr), hiddenDim,
        startTokenIdx);
}

} // namespace dev

// TODO: it would be nice to move some of that logic to Fp4Utils.h
template <tg::SfLayout Layout>
inline __device__ int32_t getSfOffset(int32_t dataRowIdx, int32_t dataBlkColIdx, int32_t numDataBlksPerRow)
{
    if constexpr (Layout == tg::SfLayout::Linear)
    {
        return numDataBlksPerRow * dataRowIdx + dataBlkColIdx;
    }
    else if constexpr (Layout == tg::SfLayout::R128c4)
    {
        return static_cast<int32_t>(dev::getSfOffset(dataRowIdx, dataBlkColIdx, numDataBlksPerRow));
    }
    else if constexpr (Layout == tg::SfLayout::R8c4 || Layout == tg::SfLayout::R8c16)
    {
        static int32_t constexpr NumRowsPerSfBlock = 8;
        static int32_t constexpr NumColsPerSfBlock = (Layout == tg::SfLayout::R8c4) ? 4 : 16;
        static int32_t constexpr NumBytesPerSfBlock = NumRowsPerSfBlock * NumColsPerSfBlock;
        int sfBlkRowIdx = dataRowIdx / NumRowsPerSfBlock;
        int sfBlkColIdx = dataBlkColIdx / NumColsPerSfBlock;
        int sfBlkIdx = sfBlkRowIdx * numDataBlksPerRow / NumColsPerSfBlock + sfBlkColIdx;
        int sfRowIdx = dataRowIdx % NumRowsPerSfBlock;
        int sfColIdx = dataBlkColIdx % NumColsPerSfBlock;
        return sfBlkIdx * NumBytesPerSfBlock + sfRowIdx * NumColsPerSfBlock + sfColIdx;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <tg::SfLayout LayoutSrc, tg::SfLayout LayoutDst, typename KernelParams>
__device__ void convertSfCommon(KernelParams params)
{
    // Note: it's assumed that the number of scaling factors per row is a multiple of 4.
    constexpr int VecSize = 4;
    using VecType = uint32_t;
    static_assert(sizeof(VecType) == VecSize);

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // Immediately trigger the secondary kernel when using PDL, then wait on primary.
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
        cudaGridDependencySynchronize();
    }
#endif

    // TODO: consider optimizing if used in production.
    // This is a naive kernel. It's not doing coalesced loads.

    int const numSfPerRow = params.hiddenDimSf;

    for (int tokenIdx = blockIdx.y; tokenIdx < params.numTokens; tokenIdx += gridDim.y)
    {
        for (int hiddenSfVecIdx = threadIdx.x + blockDim.x * blockIdx.x; hiddenSfVecIdx < numSfPerRow / VecSize;
             hiddenSfVecIdx += blockDim.x * gridDim.x)
        {
            // Index of the first SF in the vector.
            int const hiddenSfIdx = VecSize * hiddenSfVecIdx;

            // Load scale factors.
            int sfIdxIn = getSfOffset<LayoutSrc>(tokenIdx, hiddenSfIdx, numSfPerRow);
            const VecType sfVec = reinterpret_cast<VecType const*>(params.inSfPtr)[sfIdxIn / VecSize];

            // Store scale factors.
            int const sfIdxOut = getSfOffset<LayoutDst>(tokenIdx, hiddenSfIdx, numSfPerRow);
            reinterpret_cast<VecType*>(params.outSfPtr)[sfIdxOut / VecSize] = sfVec;
        }
    }
}

#define CONVERT_FP4_SF_KERNEL(LayoutSrc, LayoutDst)                                                                    \
    template <typename KernelParams>                                                                                   \
    __global__ void convertSf##LayoutSrc##To##LayoutDst##Kernel(KernelParams params)                                   \
    {                                                                                                                  \
        convertSfCommon<tg::SfLayout::LayoutSrc, tg::SfLayout::LayoutDst>(params);                                     \
    }
// We only need a conversion to the linear layout.
CONVERT_FP4_SF_KERNEL(R128c4, Linear);
CONVERT_FP4_SF_KERNEL(R8c4, Linear);
CONVERT_FP4_SF_KERNEL(R8c16, Linear);
#undef CONVERT_FP4_SF_KERNEL

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    constexpr int VecSize = 4;
    int const numThreads = 128;
    int const numBlocksX = (data.hiddenDimSf / VecSize - 1 + numThreads) / numThreads;
    int const numBlocksY = data.numTokens;
    dim3 numBlocks(numBlocksX, numBlocksY);
#define CONVERT_FP4_SF_LAUNCH(LayoutSrc, LayoutDst)                                                                    \
    if (data.sfLayoutSrc == tg::SfLayout::LayoutSrc && data.sfLayoutDst == tg::SfLayout::LayoutDst)                    \
    {                                                                                                                  \
        LAUNCH_PDL(data, false, cutlass::float_e4m3_t, convertSf##LayoutSrc##To##LayoutDst##Kernel, numBlocks,         \
            numThreads, 0, stream);                                                                                    \
        return;                                                                                                        \
    }
    CONVERT_FP4_SF_LAUNCH(R128c4, Linear);
    CONVERT_FP4_SF_LAUNCH(R8c4, Linear);
    CONVERT_FP4_SF_LAUNCH(R8c16, Linear);
#undef CONVERT_FP4_SF_LAUNCH
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace convertsf

namespace permute
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace tg = trtllm::gen;

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void permuteKernel(KernelParams params)
{
    using Type = typename KernelParams::Type;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // immediately trigger the secondary kernel when using PDL, then wait on primary
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
        cudaGridDependencySynchronize();
    }
#endif

    for (int tokenIdx = blockIdx.y; tokenIdx < params.numTokens; tokenIdx += gridDim.y)
    {
        // Loop over hidden dim
        for (int hiddenIdx = threadIdx.x + blockDim.x * blockIdx.x; hiddenIdx < params.hiddenDim;
             hiddenIdx += blockDim.x * gridDim.x)
        {

            // Load chunk of token into registers
            const Type data = params.inPtr[tokenIdx * params.hiddenDim + hiddenIdx];

            // Write to topK places
            for (int k = 0; k < params.topK; k++)
            {
                int const expandedIdx = tokenIdx * params.topK + k;
                int const permutedIdx = params.expandedIdxToPermutedIdx[expandedIdx];
                params.outPtr[permutedIdx * params.hiddenDim + hiddenIdx] = data;
            }
        }
        if (params.useDeepSeekFp8)
        {
            for (int scaleIdx = threadIdx.x + blockDim.x * blockIdx.x; scaleIdx < params.hiddenDim / 128;
                 scaleIdx += blockDim.x * gridDim.x)
            {
                for (int k = 0; k < params.topK; k++)
                {
                    int const expandedIdx = tokenIdx * params.topK + k;
                    int const permutedIdx = params.expandedIdxToPermutedIdx[expandedIdx];

                    int const idx_in = tokenIdx + params.numTokens * scaleIdx;
                    int const idx_out = permutedIdx + params.totalNumPaddedTokens[0] * scaleIdx;

                    params.outDqSfsPtr[idx_out] = params.inDqSfsPtr[idx_in];
                }
            }
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    int const numThreads = 256;
    int const numBlocksX = (data.hiddenDim - 1 + numThreads) / numThreads;
    int const numBlocksY = data.numTokens;
    dim3 numBlocks(numBlocksX, numBlocksY);

    LAUNCH(data, permuteKernel, numBlocks, numThreads, 0, stream);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace permute

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace finalize
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace tg = trtllm::gen;

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void finalizeKernel(KernelParams params)
{
    using Type = typename KernelParams::Type;
    using TypeExpW = typename KernelParams::TypeExpW;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // wait on primary kernel when using PDL
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }
#endif

    for (int tokenIdx = blockIdx.y; tokenIdx < params.numTokens; tokenIdx += gridDim.y)
    {
        // Loop over hidden dim
        for (int hiddenIdx = threadIdx.x + blockDim.x * blockIdx.x; hiddenIdx < params.hiddenDim;
             hiddenIdx += blockDim.x * gridDim.x)
        {

            // Accumulate chunk of token into registers
            float data = 0.0F;

            // Write to topK places
            for (int k = 0; k < params.topK; k++)
            {
                int const expandedIdx = tokenIdx * params.topK + k;
                const TypeExpW scale = params.expertWeightsPtr[expandedIdx];

                int const permutedIdx = params.expandedIdxToPermutedIdx[expandedIdx];
                if (permutedIdx == -1)
                    continue;
                data += float{scale} * float{params.inPtr[permutedIdx * params.hiddenDim + hiddenIdx]};
            }

            params.outPtr[tokenIdx * params.hiddenDim + hiddenIdx] = static_cast<Type>(data);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void finalizeDeepSeekKernel(KernelParams params)
{
    using Type = typename KernelParams::Type;
    using BlockReduce = hipcub::BlockReduce<float, 128>;

    __shared__ float s_scaleOut;
    __shared__ typename BlockReduce::TempStorage temp_storage;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // wait on primary kernel when using PDL
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }
#endif

    for (int tokenIdx = blockIdx.y; tokenIdx < params.numTokens; tokenIdx += gridDim.y)
    {
        // Loop over hidden dim
        for (int hiddenIdx = threadIdx.x + blockDim.x * blockIdx.x; hiddenIdx < params.hiddenDim;
             hiddenIdx += blockDim.x * gridDim.x)
        {

            // Accumulate chunk of token into registers
            float acc = 0.0f;

            for (int k = 0; k < params.topK; k++)
            {
                int const expandedIdx = tokenIdx * params.topK + k;
                int const permutedIdx = params.expandedIdxToPermutedIdx[expandedIdx];
                if (permutedIdx == -1)
                    continue;
                int const totalNumPaddedTokens = params.totalNumPaddedTokens[0];
                int const scaleIdx = permutedIdx + totalNumPaddedTokens * (hiddenIdx / 128);
                float const blockScale = params.inDqSfsPtr ? params.inDqSfsPtr[scaleIdx] : 1;

                float const expertProb = (float) params.expertWeightsPtr[tokenIdx * params.topK + k];

                float const scale = expertProb * blockScale;
                acc += scale * static_cast<float>(params.inPtr[permutedIdx * params.hiddenDim + hiddenIdx]);
            }

            // The largest (finite) value that can be represented using E4m3.
            float constexpr E4m3MaxVal{448.f};

            // Compute the absolute max
            float aMax = BlockReduce(temp_storage).Reduce(fabsf(acc), hipcub::Max());

            if (threadIdx.x == 0)
            {
                if (params.outDqSfsPtr)
                {
                    s_scaleOut = aMax / E4m3MaxVal;
                    int const scaleOut_idx = tokenIdx + hiddenIdx / 128 * params.numTokens;
                    params.outDqSfsPtr[scaleOut_idx] = aMax / E4m3MaxVal;
                }
                else
                {
                    s_scaleOut = 1.0f;
                }
            }
            __syncthreads();
            float const scaleOut = s_scaleOut;
            __syncthreads();
            params.outPtr[tokenIdx * params.hiddenDim + hiddenIdx] = (Type) (acc / scaleOut);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    if (data.mUseDeepSeekFp8)
    {
        int const numThreads = 128;
        int const numBlocksX = (data.hiddenDim - 1 + numThreads) / numThreads;
        int const numBlocksY = data.numTokens;
        dim3 numBlocks(numBlocksX, numBlocksY);

        LAUNCH_EXPW(data, finalizeDeepSeekKernel, numBlocks, numThreads, 0, stream);
    }
    else
    {
        int const numThreads = 256;
        int const numBlocksX = (data.hiddenDim - 1 + numThreads) / numThreads;
        int const numBlocksY = data.numTokens;
        dim3 numBlocks(numBlocksX, numBlocksY);

        LAUNCH_EXPW(data, finalizeKernel, numBlocks, numThreads, 0, stream);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace finalize

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace moe::dev
