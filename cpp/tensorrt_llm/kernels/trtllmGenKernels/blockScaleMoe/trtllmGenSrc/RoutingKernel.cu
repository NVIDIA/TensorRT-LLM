#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "DevKernel.h"
#include "RoutingKernel.h"
//// FIX
#include "macros.h" // #include <utils/macros.h>

#include "Utils.h"  // #include <trtllm/dev/Utils.h>

// #include "trtllmGenSrc/gen/GenCtx.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>

#include <cute/arch/cluster_sm90.hpp>

#include <type_traits>

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace moe::dev
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace routing
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace tg = trtllm::gen;
namespace cg = cooperative_groups;

////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 256;
static constexpr int NumBlocksPerCluster = 8;
static constexpr int NumThreadsGemm = 128;
static constexpr int WarpSize = 32;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int NumTopGroups = 4;
static constexpr int NumTopGroupScores = 2;
static constexpr int NumTopExperts = 8;

// Performance tuning knob.
static constexpr int NumEltsPerOffsetTilePerThread = 8;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 1000 && defined(__CUDA_ARCH_FEAT_SM100_ALL))
#define TLLM_GEN_ENABLE_FAST_REDUX
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename TypeExpW_>
struct TopKRedType
{
    using TypeExpW = TypeExpW_;
    static_assert(std::is_same_v<TypeExpW, float> || std::is_same_v<TypeExpW, cutlass::bfloat16_t>,
        "Top K reduction only implemented for float and Bf16");
    using TypeCmp = std::conditional_t<sizeof(TypeExpW) >= 4, double, float>;
    static constexpr int64_t Mask64 = 0x000000000000FFFF;
    static constexpr int32_t Mask32 = 0x0000FFFF;

    TypeCmp compVal;

    static __host__ __device__ inline TypeCmp makeCmpVal(TypeExpW val, int32_t idx = 0)
    {
        auto cmpVal = TypeCmp{val};
        TypeCmp cmpValWithIdx;
        if constexpr (sizeof(TypeExpW) >= 4)
        {
            auto cmpValIdx64 = reinterpret_cast<int64_t&>(cmpVal) | (Mask64& int64_t{idx});
            cmpValWithIdx = reinterpret_cast<TypeCmp&>(cmpValIdx64);
        }
        else
        {
            auto cmpValIdx32 = reinterpret_cast<int32_t&>(cmpVal) | (Mask32 & idx);
            cmpValWithIdx = reinterpret_cast<TypeCmp&>(cmpValIdx32);
        }
        return cmpValWithIdx;
    }

    static __host__ __device__ inline void unpack(TypeExpW& val, int32_t& idx, TypeCmp cmp)
    {
        if constexpr (sizeof(TypeExpW) >= 4)
        {
            idx = static_cast<int32_t>(reinterpret_cast<int64_t&>(cmp) & Mask64);
            auto val64 = reinterpret_cast<int64_t&>(cmp) & ~Mask64;
            val = static_cast<float>(reinterpret_cast<double&>(val64));
        }
        else
        {
            idx = reinterpret_cast<int32_t&>(cmp) & Mask32;
            auto val32 = reinterpret_cast<int32_t&>(cmp) >> 16;
            val = TypeExpW::bitcast(reinterpret_cast<uint16_t&>(val32));
        }
    }

    __host__ __device__ TopKRedType() = default;

    __host__ __device__ TopKRedType(TypeExpW val, int32_t idx)
        : compVal(makeCmpVal(val, idx))
    {
    }

    __host__ __device__ operator TypeCmp() const noexcept
    {
        return compVal;
    }

    __device__ inline TypeCmp reduce(cg::thread_block_tile<WarpSize> const& warp)
    {
#if defined(TLLM_GEN_ENABLE_FAST_REDUX)
        static constexpr bool UseCg = false;
#else
        static constexpr bool UseCg = true;
#endif
        if constexpr (UseCg || sizeof(TypeExpW) >= 4)
        {
            return cg::reduce(warp, compVal, cg::greater<TypeCmp>{});
        }
        else
        {
            float result;
            asm("redux.sync.max.f32 %0, %1, 0xffffffff;\n" : "=f"(result) : "f"(compVal));
            return result;
        }
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

static __device__ inline float tanh_fast(float x)
{
    float res;
    asm volatile("{ tanh.approx.f32 %0, %1; }\n" : "=f"(res) : "f"(x));
    return res;
}

static __device__ inline float sigmoid_fast(float x)
{
    return 0.5f * tanh_fast(0.5f * x) + 0.5f;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static __device__ inline float sigmoid_accurate(float x)
{
    return 0.5f * tanhf(0.5f * x) + 0.5f;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K_, bool Enable_>
struct TopKIdx
{
    // by default, empty
};

template <int K_>
struct TopKIdx<K_, true>
{
    static constexpr int K = K_;
    int32_t val[K];
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K, typename Type>
__device__ void reduceTopK(cg::thread_block_tile<WarpSize> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type value, int32_t idx, Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WarpSize, "Top K must have K < WarpSize");
    using RedType = TopKRedType<Type>;
    RedType topK{value, idx};
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        topK = kk > 0 && packedMax == topK.compVal ? RedType{minValue, idx} : topK;
        // get the next largest value
        packedMax = topK.reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

#define TOPK_SWAP(I, J)                                                                                                \
    {                                                                                                                  \
        auto pairMin = min(topK[I].compVal, topK[J].compVal);                                                          \
        auto pairMax = max(topK[I].compVal, topK[J].compVal);                                                          \
        topK[I].compVal = pairMax;                                                                                     \
        topK[J].compVal = pairMin;                                                                                     \
    }

template <int K, typename Type, int N, bool IsSorted = false>
__device__ void reduceTopK(cg::thread_block_tile<WarpSize> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type (&value)[N], int32_t (&idx)[N], Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WarpSize, "Top K must have K < WarpSize");
    static_assert(N > 0, "Top K must have N > 1");
    static_assert(N <= K, "Top K must have N < K");
    using RedType = TopKRedType<Type>;
    RedType topK[N];
#pragma unroll
    for (int nn = 0; nn < N; ++nn)
        topK[nn] = RedType{value[nn], idx[nn]};
    if constexpr (!IsSorted)
    {
        TOPK_SWAP(0, 2);
        TOPK_SWAP(1, 3);

        TOPK_SWAP(0, 1);
        TOPK_SWAP(2, 3);

        TOPK_SWAP(1, 2);
    }
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        bool update = kk > 0 && packedMax == topK[0].compVal;
#pragma unroll
        for (int nn = 0; nn < N; ++nn)
        {
            topK[nn] = update && nn == N - 1 ? RedType{minValue, idx[nn]} : update ? topK[nn + 1] : topK[nn];
        }
        // get the next largest value
        packedMax = topK[0].reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

#undef TOPK_SWAP

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void routingKernelGemm(KernelParams params)
{
    // naive Gemm, to be replaced by performant kernel
    using Type = typename KernelParams::Type;
    using TypeExpW = typename KernelParams::TypeExpW;
    // each thread has space for the dot product of each expert here
    extern __shared__ char __attribute((aligned(128))) smemBase[];
    auto* smemDotPartial = reinterpret_cast<float*>(smemBase);
    static constexpr int SmemStride = NumThreadsGemm + 1;

    auto tokenOff = int64_t{blockIdx.x} * int64_t{params.mHiddenDim};

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // immediately trigger the secondary kernel when using PDL
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
#endif

    // dot product for all experts
    // entire block must go into this loop
    for (int32_t dd = threadIdx.x; dd < params.mHiddenDim; dd += NumThreadsGemm)
    {
        Type act = params.mPtrIn[tokenOff + dd];

        for (int32_t expertIdx = 0; expertIdx < params.mNumExperts; ++expertIdx)
        {
            auto weightOff = int64_t{expertIdx} * int64_t{params.mHiddenDim};
            TypeExpW weight = params.mPtrRoutingWeights[weightOff + dd];
            auto val = float{act} * float{weight};
            if (dd == threadIdx.x)
            {
                smemDotPartial[expertIdx * SmemStride + threadIdx.x] = val;
            }
            else
            {
                smemDotPartial[expertIdx * SmemStride + threadIdx.x] += val;
            }
        }
    }
    // make all partial dot products available to all threads
    __syncthreads();

    // finalize dot product and write to output
    for (int32_t expertIdx = threadIdx.x; expertIdx < params.mNumExperts; expertIdx += NumThreadsGemm)
    {
        float dot = 0.F;
        for (int32_t ii = 0; ii < NumThreadsGemm; ++ii)
        {
            dot += smemDotPartial[expertIdx * SmemStride + ii];
        }
        params.mPtrScores[int64_t{blockIdx.x} * int64_t{params.mNumExperts} + expertIdx] = dot;
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T mulLog2(T a, T bLog2)
{
    return a << bLog2;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T divUpLog2(T a, T bLog2)
{
    return ((a + (1 << bLog2) - 1) >> bLog2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T divUpMulLog2(T a, T bLog2)
{
    return mulLog2<T>(divUpLog2<T>(a, bLog2), bLog2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void routingMainKernel(KernelParams params)
{
    // declare types required for reductions
    using TypeExpW = typename KernelParams::TypeExpW;

    // declare shared memory structure
    // number of experts is bounded by number of threads
    __shared__ float __attribute((aligned(128))) smemScoreSigmoid[NumThreads];
    __shared__ float __attribute((aligned(128))) smemScoreBias[NumThreads];
    // number of expert groups is bounded by number of warps
    __shared__ float __attribute((aligned(128))) smemGroupScores[NumWarps];

    // needed for warp reduce
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);
    // for the final reduction of weight norm, only some lanes need to participate
    int32_t laneIdx = threadIdx.x % WarpSize;
    int32_t warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
    // warps outside the range of expert groups do not participate
    if (warpIdx >= params.mNumExpertGroups)
    {
        return;
    }

    // note that for invalid scores, we simply use a negative value:
    // they work well even with the compacted format used in topK, and
    // sigmoid / bias activated scores cannot be negative
    static constexpr float invalidScoreFloat = -1.F;
    const TypeExpW invalidScore = TypeExpW{invalidScoreFloat};

    // load bias already; each warp represents one expert group
    auto threadExpert = warpIdx * params.mNumExpertsPerGroup + laneIdx;
    auto expertSelected = laneIdx < params.mNumExpertsPerGroup;
    auto scoreIdx = int64_t{blockIdx.x} * int64_t{params.mNumExperts} + threadExpert;
    auto biasVal = expertSelected ? params.mPtrRoutingBias[threadExpert] : invalidScore;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // trigger the secondary kernel when using PDL, then wait on primary
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
        cudaGridDependencySynchronize();
    }
#endif

    // get our assigned thread score; each warp represents one expert group
    float score = expertSelected ? params.mPtrScores[scoreIdx] : invalidScoreFloat;
    // get the sigmoid score
    // note that for invalid values, we simply use a negative value:
    // sigmoig scores are always strictly positive
    auto scoreSigmoid = sigmoid_accurate(score);
    // write the sigmoid score to shared for later use
    if (expertSelected)
    {
        smemScoreSigmoid[threadExpert] = scoreSigmoid;
    }
    // get the score with bias
    // note that with invalid values, because sigmoid is < 1 and bias is -1,
    // we must get a negative value, which is smaller than any valid value
    // TODO: verify bf16 scoreBias accuracy before changing it back to bf16
    // auto scoreBias = TypeExpW{scoreSigmoid + float{biasVal}}; // TypeExpW is bf16
    auto scoreBias = float{scoreSigmoid + float{biasVal}};
    if (expertSelected)
    {
        smemScoreBias[threadExpert] = scoreBias;
    }

    // registers for top group score reduction
    float topExpGroupScores[NumTopGroupScores];
    [[maybe_unused]] int32_t topExpGroupIdx[NumTopGroupScores];
    reduceTopK(warp, topExpGroupScores, topExpGroupIdx, scoreBias, threadExpert,
        /* minValue */ invalidScoreFloat);

    // get the final group score and write it to shared
    if (cute::elect_one_sync())
    {
        auto groupScore = topExpGroupScores[0] + topExpGroupScores[1];
        smemGroupScores[warpIdx] = groupScore;
    }

    // make group scores available to all warps
    __syncthreads();

    float topGroups[NumTopGroups]; // params.mNumLimitedGroups
    int32_t topGroupIdx[NumTopGroups];
    float expertScoreGroup[NumTopGroups];
    int32_t expertIdxGroup[NumTopGroups];
    float topScores[NumTopExperts]; // params.mTopK
    int32_t topExperts[NumTopExperts];
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    if (warpIdx == 0)
    {
        // a single warp performs the selection of top groups, and goes on to select the final experts
        float groupScore = laneIdx < params.mNumExpertGroups ? smemGroupScores[laneIdx] : float{};

        reduceTopK(warp, topGroups, topGroupIdx, groupScore, laneIdx,
            /* minValue */ invalidScoreFloat);

        // final expert selection: get relevant indexes and scores from shared

#pragma unroll
        for (int ii = 0; ii < NumTopGroups; ++ii)
        { // params.mNumLimitedGroups
            auto groupIdx = topGroupIdx[ii];
            expertIdxGroup[ii] = groupIdx * params.mNumExpertsPerGroup + laneIdx;
            expertScoreGroup[ii] = expertSelected ? smemScoreBias[expertIdxGroup[ii]] : invalidScoreFloat;
        }

        reduceTopK(warp, topScores, topExperts, expertScoreGroup, expertIdxGroup,
            /* minValue */ invalidScoreFloat);

        // determine our lane's expert index and write to output
        int32_t expertIdx = 0;
#pragma unroll
        for (int ii = 0; ii < NumTopExperts; ++ii)
        { // params.mTopK
            expertIdx = laneIdx == ii ? topExperts[ii] : expertIdx;
        }
        // determine whether our expert is local to this GPU
        auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;

        // write expert idx out already
        auto idxTopK = blockIdx.x * NumTopExperts + laneIdx; // params.mTopK
        if (laneIdx < NumTopExperts && params.mPtrExpertIdx != nullptr)
        {                                                    // params.mTopK
            params.mPtrExpertIdx[idxTopK] = expertIdx;
        }
        float scoreNorm = laneIdx < NumTopExperts ? smemScoreSigmoid[expertIdx] : 0.F;
        auto redNorm = cg::reduce(warp, scoreNorm, cg::plus<float>{});
        auto finalScore = TypeExpW{scoreNorm * params.mRouteScale / redNorm};
        if (laneIdx < NumTopExperts && params.mPtrExpertWeights != nullptr)
        { // params.mTopK
            params.mPtrExpertWeights[idxTopK] = finalScore;
        }
        if (laneIdx < NumTopExperts && params.mPtrExpertWeightsFull != nullptr && isLocalExpert)
        { // params.mTopK
            auto idxWeightsFull = localExpertIdx * gridDim.x + blockIdx.x;
            params.mPtrExpertWeightsFull[idxWeightsFull] = finalScore;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params)
{
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
    // needed for the exclusive sum of token offsets
    using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    // Number of threads in the cluster.
    static constexpr int NumThreadsPerCluster = NumThreads * NumBlocksPerCluster;
    // If the number of tokens is bounded by 16384, then the total number of indexes
    // is bounded by 16384 * TopK.
    // TODO: if we only use this kernel up to 1024 tokens, we could use 1024 here.
    static constexpr int MaxExpandedIdxPerThread
        = (16384 * NumTopExperts + NumThreadsPerCluster - 1) / NumThreadsPerCluster;

    // Initialize cluster.
    uint32_t const clusterBlockRank = blockIdx.x;
    uint32_t const clusterThreadIdx = NumThreads * clusterBlockRank + threadIdx.x;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

    auto expandedIdxSize = params.mNumTokens * NumTopExperts;

    // pre-fill the counts with 0
    smemExpertCount[threadIdx.x] = 0;
    __syncthreads();

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    // each thread keeps has some number of "expanded indexes" assigned to it
    // for each of these, we keep the associated expert and offset within expert in registers
    int32_t expertIndexes[MaxExpandedIdxPerThread];
    int32_t expertOffsets[MaxExpandedIdxPerThread];
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
    // time, and branch between a fast path without bound checks and a slow path with bound checks.
    int constexpr IterStride = 4;
    static_assert(MaxExpandedIdxPerThread % IterStride == 0);

    // Define a lambda to avoid code duplication in both branches.
    auto loopBody = [&](int ii, int expandedIdx)
    {
        int32_t expertIdx = params.mPtrExpertIdx[expandedIdx];
        expertIndexes[ii] = expertIdx;
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + expertIdx, 1) : 0;
    };

#pragma unroll
    for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride)
    {
        // Whether it's safe to do multiple iterations without bound checks.
        bool const takeFastPath = (ii0 + IterStride) * NumThreadsPerCluster <= expandedIdxSize;
        if (takeFastPath)
        {
#pragma unroll
            for (int32_t jj = 0; jj < IterStride; jj++)
            {
                int const ii = ii0 + jj;
                auto expandedIdx = static_cast<int32_t>(clusterThreadIdx) + ii * NumThreadsPerCluster;
                loopBody(ii, expandedIdx);
            }
        }
        else
        {
            bool doBreak = false;
#pragma unroll
            for (int32_t jj = 0; jj < IterStride; jj++)
            {
                int const ii = ii0 + jj;
                auto expandedIdx = static_cast<int32_t>(clusterThreadIdx) + ii * NumThreadsPerCluster;
                if (expandedIdx >= expandedIdxSize)
                {
                    doBreak = true;
                    break;
                }
                loopBody(ii, expandedIdx);
            }
            if (doBreak)
            {
                break;
            }
        }
    }

    // Make local histogram (token counts per expert) available to all threads in the cluster.
    cg::cluster_group::sync();

    //
    // Each thread now represents one expert
    //

    // Get the histogram bin from each rank for this expert.
    int32_t expertCounts[NumBlocksPerCluster];
#pragma unroll
    for (int rank = 0; rank < NumBlocksPerCluster; rank++)
    {
        int32_t const* remoteSmem = cg::cluster_group::map_shared_rank(smemExpertCount, rank);
        expertCounts[rank] = remoteSmem[threadIdx.x];
    }

    // Compute an exclusive prefix sum of the block-local count.
    // Each block only needs the count up to its rank, and the total count.
    int32_t count = 0;
    int32_t blockExpertOffset = 0;
#pragma unroll
    for (int rank = 0; rank < NumBlocksPerCluster; rank++)
    {
        if (rank == clusterBlockRank)
        {
            blockExpertOffset = count;
        }
        count += expertCounts[rank];
    }

    // Arrive: we do not access distributed shared memory after this point.
    __cluster_barrier_arrive();

    // Compute the runtime config for projections
    // Whether or not an expert is local is taken into account when smemExpertCount is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    // Strided loop to share this work between blocks.
    int32_t tokensPerTile = params.mAllToAllRouteAct ? params.mNumTokens : count;
    for (int32_t cta = clusterBlockRank; cta < numCta; cta += NumBlocksPerCluster)
    {
        const int32_t localExpertIdx = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
        params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
        params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta] = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
            mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + tokensPerTile);
    }

    // get the padded offset associated with this expert
    const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
    const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);

    // write out padded count
    if (clusterBlockRank == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync())
    {
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    // write expert offsets to shared
    smemExpertOffset[threadIdx.x] = offset + blockExpertOffset;

    // make expert offsets available to all threads
    __syncthreads();

    // Wait: we cannot exit while other blocks may be accessing the current block's shared memory.
    // Note (lsugy): I observed a perf benefit to doing this before the final loop so the compiler can
    // implement break with EXIT.
    __cluster_barrier_wait();

    // trigger the secondary kernel when using PDL
    // We can't do it earlier because FC1 depends on the mPtrCtaIdxXyToBatchIdx,
    // mPtrCtaIdxXyToMnLimit, mPtrNumNonExitingCtas and mPtrTotalNumPaddedTokens
    // TODO: this is not sufficient to ensure visibility in the next kernel!

    // TODO: disable PDL for now to avoid race condition in FC1
    if constexpr (KernelParams::UsePdl)
    {
        // cudaTriggerProgrammaticLaunchCompletion();
    }

// each thread has the same "expanded indexes" assigned to it as above
// at this point, we know the final offsets of experts and the offsets within
// experts, which allows writing the final index values
#pragma unroll
    for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ++ii)
    {
        auto expandedIdx = static_cast<int32_t>(clusterThreadIdx) + ii * NumThreadsPerCluster;
        if (expandedIdx >= expandedIdxSize)
        {
            break;
        }
        auto expertIdx = expertIndexes[ii];
        // check whether this expert is local to our GPU at all
        auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        auto tokenIdx = expandedIdx / NumTopExperts;
        auto permutedIdx = isLocalExpert ? int32_t{smemExpertOffset[expertIdx]} + expertOffsets[ii] : int32_t{-1};
        if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
        {
            params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
        }
        if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
        {
            params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
        }
    }
}
#else
__global__ void routingIndicesClusterKernel(KernelParams params)
{
    assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif
////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void __launch_bounds__(NumThreads) routingIndicesCoopKernel(KernelParams params)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
    // needed for the exclusive sum of token offsets
    using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    // 64 elements -> 128+ registers. Above that we may start to see spilling to local memory.
    static constexpr int MaxExpandedIdxPerThread = 64;

    // Initialize grid.
    cg::grid_group grid = cg::this_grid();
    // Note: the following is more efficient than grid.block_index() because we don't use y and z.
    uint32_t const gridBlockIdx = blockIdx.x;
    uint32_t const gridThreadIdx = NumThreads * gridBlockIdx + threadIdx.x;
    uint32_t const numBlocks = gridDim.x;
    uint32_t const numThreadsPerGrid = numBlocks * NumThreads;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

    auto expandedIdxSize = params.mNumTokens * NumTopExperts;

    // pre-fill the counts with 0
    smemExpertCount[threadIdx.x] = 0;
    __syncthreads();

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    // each thread keeps has some number of "expanded indexes" assigned to it
    // for each of these, we keep the associated expert and offset within expert in registers
    int32_t expertIndexes[MaxExpandedIdxPerThread];
    int32_t expertOffsets[MaxExpandedIdxPerThread];
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
    // time, and branch between a fast path without bound checks and a slow path with bound checks.
    int constexpr IterStride = 4;
    static_assert(MaxExpandedIdxPerThread % IterStride == 0);

    // Define a lambda to avoid code duplication in both branches.
    auto loopBody = [&](int ii, int expandedIdx)
    {
        int32_t expertIdx = params.mPtrExpertIdx[expandedIdx];
        expertIndexes[ii] = expertIdx;
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + expertIdx, 1) : 0;
    };

#pragma unroll
    for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride)
    {
        // Whether it's safe to do multiple iterations without bound checks.
        bool const takeFastPath = (ii0 + IterStride) * numThreadsPerGrid <= expandedIdxSize;
        if (takeFastPath)
        {
#pragma unroll
            for (int32_t jj = 0; jj < IterStride; jj++)
            {
                int const ii = ii0 + jj;
                auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
                loopBody(ii, expandedIdx);
            }
        }
        else
        {
            bool doBreak = false;
#pragma unroll
            for (int32_t jj = 0; jj < IterStride; jj++)
            {
                int const ii = ii0 + jj;
                auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
                if (expandedIdx >= expandedIdxSize)
                {
                    doBreak = true;
                    break;
                }
                loopBody(ii, expandedIdx);
            }
            if (doBreak)
            {
                break;
            }
        }
    }

    // Make histogram (token counts per expert) available to all threads in the block.
    __syncthreads();

    //
    // Each thread now represents one expert
    //

    // Add the local bin count to the common bin count and get a per-CTA offset.
    int32_t const localExpertCount = smemExpertCount[threadIdx.x];
    int32_t const blockExpertOffset = atomicAdd(&params.mPtrExpertCounts[threadIdx.x], localExpertCount);

    // Sync to wait for completion of the histogram reduction.
    grid.sync();

    // Get total count for this expert.
    int32_t count = params.mPtrExpertCounts[threadIdx.x];

    // Note: the scan is redundant in all CTAs, but doing it in only 1 CTA would be worse for latency.

    // Compute the runtime config for projections
    // Whether or not an expert is local is taken into account when smemExpertCount is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    // Strided loop to share this work between blocks.
    int32_t tokensPerTile = params.mAllToAllRouteAct ? params.mNumTokens : count;
    for (int32_t cta = gridBlockIdx; cta < numCta; cta += numBlocks)
    {
        const int32_t localExpertIdx = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
        params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
        params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta] = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
            mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + tokensPerTile);
    }

    // get the padded offset associated with this expert
    const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
    const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);

    // write out padded count
    if (gridBlockIdx == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync())
    {
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    // write expert offsets to shared
    smemExpertOffset[threadIdx.x] = offset + blockExpertOffset;

    // make expert offsets available to all threads
    __syncthreads();

    // trigger the secondary kernel when using PDL
    // We can't do it earlier because FC1 depends on the mPtrCtaIdxXyToBatchIdx,
    // mPtrCtaIdxXyToMnLimit, mPtrNumNonExitingCtas and mPtrTotalNumPaddedTokens
    // TODO: this is not sufficient to ensure visibility in the next kernel!
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }

// each thread has the same "expanded indexes" assigned to it as above
// at this point, we know the final offsets of experts and the offsets within
// experts, which allows writing the final index values
#pragma unroll
    for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ++ii)
    {
        auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
        if (expandedIdx >= expandedIdxSize)
        {
            break;
        }
        auto expertIdx = expertIndexes[ii];
        // check whether this expert is local to our GPU at all
        auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        auto tokenIdx = expandedIdx / NumTopExperts;
        auto permutedIdx = isLocalExpert ? int32_t{smemExpertOffset[expertIdx]} + expertOffsets[ii] : int32_t{-1};
        if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
        {
            params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
        }
        if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
        {
            params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
        }
    }
#else
    assert(false && "routingIndicesCoopKernel is only supported on SM90+ architectures");
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Two-step approach (if number of tokens exceed limits of what cluster / cooperative launch
// variants can handle): in order to minimize the amount of data to exchange through global memory,
// we will compute the local histograms in smem twice: the first kernel will get us the total number
// of tokens per expert. The second kernel will use the smem and L2 atomics to get corresponding
// element and tile offsets.
//
// Note: the histogram calculation could also be fused with routingMainKernel, but this might be
// inefficient if we have one CTA per token doing a single global atomic.

template <typename KernelParams>
__global__ void __launch_bounds__(NumThreads) routingIndicesHistogramKernel(KernelParams params)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];

    // For unrolling.
    uint32_t constexpr NumEltsPerThread = 8;

    // Pre-fill the counts with 0
    smemExpertCount[threadIdx.x] = 0;
    __syncthreads();

    // Wait on primary grid and trigger secondary kernel.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
        cudaTriggerProgrammaticLaunchCompletion();
    }

    uint32_t const expandedIdxSize = params.mNumTokens * NumTopExperts;
    uint32_t const localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;

    uint32_t const gridBlockOffset = blockIdx.x * NumThreads;
    uint32_t const gridStride = gridDim.x * NumThreads;

    // Define a lambda to avoid code duplication in branches.
    auto loopBody = [&](int expandedIdx)
    {
        int32_t expertIdx = params.mPtrExpertIdx[expandedIdx];
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        if (isLocalExpert)
        {
            atomicAdd(&smemExpertCount[expertIdx], 1);
        }
    };

    // Grid-stride loop.
    for (uint32_t expandedIdx0 = gridBlockOffset * NumEltsPerThread; expandedIdx0 < expandedIdxSize;
         expandedIdx0 += gridStride * NumEltsPerThread)
    {
        // Fast path if bound checks aren't necessary
        if (expandedIdx0 + NumEltsPerThread * NumThreads <= expandedIdxSize)
        {
#pragma unroll
            for (uint32_t ii = 0; ii < NumEltsPerThread; ii++)
            {
                uint32_t expandedIdx = expandedIdx0 + ii * NumThreads + threadIdx.x;
                loopBody(expandedIdx);
            }
        }
        else
        {
            for (uint32_t expandedIdx = expandedIdx0 + threadIdx.x; expandedIdx < expandedIdxSize;
                 expandedIdx += NumThreads)
            {
                loopBody(expandedIdx);
            }
        }
    }
    __syncthreads();

    //
    // Each thread now represents one expert
    //

    // Reduce histograms with atomics.
    int32_t const localExpertCount = smemExpertCount[threadIdx.x];
    atomicAdd(&params.mPtrExpertCounts[threadIdx.x], localExpertCount);
#else
    assert(false && "routingIndicesHistogramKernel is only supported on SM90+ architectures");
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void __launch_bounds__(NumThreads) routingIndicesOffsetsKernel(KernelParams params)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertTileOffset[NumThreads];
    // needed for the exclusive sum of token offsets
    using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    static constexpr int MaxExpandedIdxPerThread = NumEltsPerOffsetTilePerThread;
    static constexpr int MaxExpandedIdxPerBlock = NumThreads * MaxExpandedIdxPerThread;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

    uint32_t const expandedIdxSize = params.mNumTokens * NumTopExperts;
    uint32_t const numTiles = (expandedIdxSize + MaxExpandedIdxPerBlock - 1) / (MaxExpandedIdxPerBlock);

    // Wait on primary grid.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    // The expert offsets are common to all tiles of all blocks.
    // Load the histogram, scan it and write offsets to shared memory.
    // Note: the scan is redundant in all CTAs. Would it make sense to use an intermediate kernel for
    // the scan, with PDL?

    // Each thread represents one expert. Get total count for this expert.
    int32_t count = params.mPtrExpertCounts[threadIdx.x];

    // Compute the runtime config for projections
    // Whether or not an expert is local is taken into account when the histogram is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    // Get the padded offset associated with this expert
    const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
    const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);

    // Write expert offsets to shared
    smemExpertOffset[threadIdx.x] = offset;
    // Sync to make expert offsets available to all threads.
    __syncthreads();

    // The first block writes out padded count
    if (blockIdx.x == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync())
    {
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    // Strided loop to share this work between blocks.
    int32_t tokensPerTile = params.mAllToAllRouteAct ? params.mNumTokens : count;
    for (int32_t cta = blockIdx.x; cta < numCta; cta += gridDim.x)
    {
        const int32_t localExpertIdx = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
        params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
        params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta] = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
            mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + tokensPerTile);
    }

    //
    // Now loop on indices and compute offsets.
    //

    // Grid-stride loop on 1D "tiles" of input indices.
    for (uint32_t tileIdx = blockIdx.x; tileIdx < numTiles; tileIdx += gridDim.x)
    {
        if (tileIdx > 0)
        {
            // Sync for safe reuse of smem buffers.
            __syncthreads();
        }

        // Pre-fill the counts with 0
        smemExpertCount[threadIdx.x] = 0;
        __syncthreads();

        // each thread keeps has some number of "expanded indexes" assigned to it
        // for each of these, we keep the associated expert and offset within expert in registers
        int32_t expertIndexes[MaxExpandedIdxPerThread];
        int32_t expertOffsets[MaxExpandedIdxPerThread];
        auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;

        // Define a lambda to avoid code duplication in branches.
        auto loopBody = [&](int ii, int expandedIdx)
        {
            int32_t expertIdx = params.mPtrExpertIdx[expandedIdx];
            expertIndexes[ii] = expertIdx;
            // check whether this expert is local to our GPU at all and ignore if not
            auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + expertIdx, 1) : 0;
        };

        // For all tiles but the last, all indices are in bounds.
        if (tileIdx < numTiles - 1)
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                loopBody(ii, expandedIdx);
            }
        }
        else
        {
            // For the last tile, we need to exit the loop when out of bounds.
            // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
            // time, and branch between a fast path without bound checks and a slow path with bound checks
            int constexpr IterStride = 4;
            static_assert(MaxExpandedIdxPerThread % IterStride == 0);

#pragma unroll
            for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride)
            {
                // Whether it's safe to do multiple iterations without bound checks.
                bool const takeFastPath
                    = tileIdx * MaxExpandedIdxPerBlock + (ii0 + IterStride) * NumThreads <= expandedIdxSize;
                if (takeFastPath)
                {
#pragma unroll
                    for (int32_t jj = 0; jj < IterStride; jj++)
                    {
                        int const ii = ii0 + jj;
                        auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                        loopBody(ii, expandedIdx);
                    }
                }
                else
                {
                    bool doBreak = false;
#pragma unroll
                    for (int32_t jj = 0; jj < IterStride; jj++)
                    {
                        int const ii = ii0 + jj;
                        auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                        if (expandedIdx >= expandedIdxSize)
                        {
                            doBreak = true;
                            break;
                        }
                        loopBody(ii, expandedIdx);
                    }
                    if (doBreak)
                    {
                        break;
                    }
                }
            }
        }

        // Make local histogram (token counts per expert) available to all threads in the block.
        __syncthreads();

        // Each thread now represents one expert

        // Add the local bin count to the common bin count and get a per-CTA offset. We use the second
        // half of the histogram buffer for this histogram, because the first half already holds the
        // reduced histogram from the previous kernel.
        int32_t const localExpertCount = smemExpertCount[threadIdx.x];
        int32_t const tileExpertOffset
            = atomicAdd(&params.mPtrExpertCounts[NumThreads + threadIdx.x], localExpertCount);

        // Make per-expert tile offsets available to all threads in the block.
        smemExpertTileOffset[threadIdx.x] = tileExpertOffset + smemExpertOffset[threadIdx.x];
        __syncthreads();

        // Add tile offset and element offset and write to global memory.
        auto storeLoopBody = [&](int ii, int expandedIdx)
        {
            int32_t expertIdx = expertIndexes[ii];
            // check whether this expert is local to our GPU at all
            auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            auto tokenIdx = expandedIdx / NumTopExperts;
            auto permutedIdx = isLocalExpert ? (expertOffsets[ii] + smemExpertTileOffset[expertIdx]) : int32_t{-1};
            if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
            {
                params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
            }
            if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
            {
                params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
            }
        };
        // Bound checks only in last tile.
        if (tileIdx < numTiles - 1)
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                storeLoopBody(ii, expandedIdx);
            }
        }
        else
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                if (expandedIdx >= expandedIdxSize)
                {
                    break;
                }
                storeLoopBody(ii, expandedIdx);
            }
        }
    }

    // Trigger secondary kernel.
    // Note: this does not guarantee the visibility of prior writes unless the consumer executes a
    // dependency sync.
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
#else
    assert(false && "routingIndicesOffsetsKernel is only supported on SM90+ architectures");
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    TLLM_CHECK_ERROR(data.mPtrExpertIdx != nullptr || data.mPtrPermutedIdxSize != nullptr
            || data.mPtrExpertWeightsFull != nullptr || data.mPtrExpertWeights != nullptr,
        "Routing kernel requires at least one output parameter");
    if (data.mPtrExpandedIdxToPermutedIdx != nullptr || data.mPtrPermutedIdxToTokenIdx != nullptr)
        TLLM_CHECK_ERROR(data.mPtrExpertIdx != nullptr && data.mPtrPermutedIdxSize,
            "If permuted index is required, `mPtrExpertIdx` is also required");
    TLLM_CHECK_ERROR(!data.mUseRoutingSoftmax, "Routing with softmax not implemented yet");
    TLLM_CHECK_ERROR(
        data.mNumLimitedGroups == NumTopGroups, "Routing kernel expects ", NumTopGroups, " groups (for now)");
    TLLM_CHECK_ERROR(data.mTopK == NumTopExperts, "Routing kernel expects ", NumTopExperts, " topK experts (for now)");
    TLLM_CHECK_ERROR(data.mTopK <= WarpSize, "Routing kernel expects top K <= warp size, got ", data.mTopK);
    TLLM_CHECK_ERROR(data.mTopK * data.mNumLimitedGroups <= WarpSize,
        "Routing kernel expects top K * top groups <= warp size (for now), got ", data.mTopK, " * ",
        data.mNumLimitedGroups);
    TLLM_CHECK_ERROR(data.mNumExperts >= NumTopExperts, "Routing kernel expects ", NumTopExperts,
        " to be at most #experts ", data.mNumExperts);
    TLLM_CHECK_ERROR(data.mNumExperts <= NumThreads, "Routing kernel expects #experts ", data.mNumExperts,
        " <= #threads ", NumThreads);
    TLLM_CHECK_ERROR(data.mNumExpertGroups <= NumWarps, "Routing kernel expects #experts groups ",
        data.mNumExpertGroups, " to be <= #warps", NumWarps);
    TLLM_CHECK_ERROR(data.mNumExperts % data.mNumExpertGroups == 0, "Routing kernel expects #experts ",
        data.mNumExperts, " to be a multiple of #expert groups ", data.mNumExpertGroups);
    TLLM_CHECK_ERROR(data.mNumExperts / data.mNumExpertGroups <= WarpSize,
        "Routing kernel expects #experts per group <= warp size, got ", data.mNumExperts / data.mNumExpertGroups);
    TLLM_CHECK_ERROR(
        data.mNumExperts % 4 == 0, "Routing kernel expects #experts ", data.mNumExperts, " to be a multiple of 4.");
    TLLM_CHECK_ERROR(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got ", data.mPaddingLog2);
    int const numBlocks = data.mNumTokens;

    if (data.mPtrExpertWeightsFull != nullptr)
    {
        auto localExpertExtent = data.mNumLocalExperts << data.mLocalExpertsStrideLog2;
        // note: we set a value of 0 here, s.t. even if the routing happens,
        // it will be ignored / not given any weight
        TLLM_CHECK_CUDA(hipMemsetAsync(
            data.mPtrExpertWeightsFull, 0, localExpertExtent * data.mNumTokens * sizeof(float), (hipStream_t) stream));
    }

    /*  disable memset(-1) for permuted_idx_to_token_idx for performance
        if (data.mPtrPermutedIdxToTokenIdx != nullptr)
        {
            // need to set all values to -1 before running the kernel
            auto maxPermutedSize
                = data.mNumTokens * data.mTopK + (data.mNumExperts << data.mPaddingLog2) - data.mNumExperts;
            // note that a value of -1 per byte works for any size of signed integer
            // to set each full value to the logical value -1
            TLLM_CHECK_CUDA(hipMemsetAsync(data.mPtrPermutedIdxToTokenIdx, -1,
                static_cast<size_t>(maxPermutedSize) * sizeof(int32_t), (hipStream_t) stream));
        }
    */

    bool const useSingleCluster = data.mNumTokens <= 1024;
    if (!useSingleCluster)
    {
        // Reset the global histograms (not used in single-cluster code path).
        // Cover both for the cooperative and two-kernel code paths.
        TLLM_CHECK_CUDA(hipMemsetAsync(
            data.mPtrExpertCounts, 0, static_cast<size_t>(2 * NumThreads) * sizeof(int32_t), (hipStream_t) stream));
    }

    // Number of blocks we can use in the cooperative kernel
    // The number of blocks must be:
    //   >= ⌈(numTokens * NumTopExperts) / (MaxExpandedIdxPerThread * NumThreads)⌉
    //   <= numSms, assuming an occupancy of 1 block/SM
    //
    // If too small for the given numTokens, fall back to the less performant two-step method.
    //
    // The upper bound is a strict requirement. The number of blocks should be determined by querying
    // the device properties, or conservatively low.
    // /!\ The following number is not portable!! (but works on H100 and B200)
    int const numBlocksCoop = 128;

    // Maximum number of tokens supported by the kernel using a cooperative launch.
    int const maxTokensCoop = (numBlocksCoop * NumThreads * 64) / NumTopExperts;
    LAUNCH_EXPW_ONLY(data,
        /*coopLaunch=*/false, routingMainKernel, numBlocks, NumThreads,
        /*smemSize=*/0, // No dynamic smem
        stream);

    if (data.mPtrPermutedIdxSize != nullptr)
    {
        if (useSingleCluster)
        {
            LAUNCH_EXPW_ONLY(data,
                /*coopLaunch=*/false, routingIndicesClusterKernel, NumBlocksPerCluster, NumThreads,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
        else if (data.mNumTokens <= maxTokensCoop)
        {
            LAUNCH_EXPW_ONLY(data,
                /*coopLaunch=*/true, routingIndicesCoopKernel, numBlocksCoop, NumThreads,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
        else
        {
            const uint32_t expandedIdxSize = data.mNumTokens * NumTopExperts;

            const uint32_t histogramEltsPerBlock = 8 * NumThreads;
            const uint32_t offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreads;

            // Limit grid size (both kernels use a grid-stride loop).
            const uint32_t maxNumBlocks = 1024;

            int const numBlocksHistogram
                = std::min((expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
            int const numBlocksOffsets
                = std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

            LAUNCH_EXPW_ONLY(data,
                /*coopLaunch=*/false, routingIndicesHistogramKernel, numBlocksHistogram, NumThreads,
                /*smemSize=*/0, // No dynamic smem
                stream);
            LAUNCH_EXPW_ONLY(data,
                /*coopLaunch=*/false, routingIndicesOffsetsKernel, numBlocksOffsets, NumThreads,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace routing

} // namespace moe::dev
