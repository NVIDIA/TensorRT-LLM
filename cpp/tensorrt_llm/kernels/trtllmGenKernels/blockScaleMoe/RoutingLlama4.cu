#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "RoutingKernel.cuh"

namespace moe::dev::routing
{
namespace routingLlama4
{

////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 1024;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int MaxNumTopExperts = 1;
static constexpr int MaxNumExperts = 128;
static constexpr int MaxNumTokensSingleCluster = NumBlocksPerCluster * NumThreads;
static constexpr int MaxNumTokensSingleClusterScores = NumBlocksPerCluster * NumWarps;
static constexpr int WarpKernelSmemStride = 33;
// with further optimization to `routingIndicesWarpKernel`, this limit may
// increase. For now, it is a good cut-off point for when the block-wise
// operations are more efficient end-to-end.
static constexpr int WarpKernelMaxNumTokens = 4;

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename DataType, int VecSize>
__forceinline__ __device__ void routingTopKExperts(cg::thread_block_tile<WarpSize> const& warp,
    DataType (&warpMaxScore)[MaxNumTopExperts], int32_t (&warpMaxExpertIdx)[MaxNumTopExperts], int32_t const laneIdx,
    int32_t const numExperts, DataType const* ptrScores)
{
    DataType minScore = DataType{-INFINITY};
    DataType maxScore = minScore;
    int32_t maxExpertIdx{-1};
    using DataTypeVec = std::conditional_t<sizeof(DataType) == 2, float2, float4>;

    // Non-vectorized loading: directly access ptrScores with expertIdx
    for (int i = 0; i < VecSize; ++i)
    {
        auto expertIdx = i * WarpSize + laneIdx;
        auto newScore = expertIdx < numExperts ? ptrScores[expertIdx] : minScore;
        // note: use `>=` s.t. highest index always wins, just like in `reduceTopK`
        if (newScore > maxScore)
        {
            maxScore = newScore;
            maxExpertIdx = expertIdx;
        }
    }

    topk::reduceTopK(warp, warpMaxScore, warpMaxExpertIdx, maxScore, maxExpertIdx, minScore);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void __launch_bounds__(WarpSize) routingIndicesWarpKernel(KernelParams params)
{
    // types used in this kernel
    using OutputT = typename KernelParams::OutputT;
    using InputT = typename KernelParams::InputT;
    using TypePacked = PackedScoreIdx<OutputT>;
    // use the default cub warp-scan, with shfl
    using Scan = hipcub::WarpScan<int32_t>;
    __shared__ typename Scan::TempStorage tempStorage;

    // each thread encodes 4 experts in one `int32_t`. The assumption is that
    // we don't have more than 127 tokens, but `WarpKernelMaxNumTokens` must be
    // smaller than that because other approaches will be more efficient for
    // 127 tokens.
    static constexpr int ExpertsPerThread = sizeof(int32_t);
    static_assert(WarpKernelMaxNumTokens <= 127);
    // this is a full table of which token is routed to which expert.
    // the assumption here is that there are no more than 128 experts.
    // we use a stride of 33 instead of 32 to avoid shared memory bank conflicts.
    __shared__ int32_t __attribute((aligned(128)))
    smemExpertTokenCountFull[WarpKernelMaxNumTokens][WarpKernelSmemStride];
    static_assert(WarpKernelSmemStride == WarpSize + 1);
    static_assert(MaxNumExperts / sizeof(int32_t) <= WarpSize);

    // values needed for the top-1 reduction, if required
    InputT minScore = InputT{-INFINITY};
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

#pragma unroll
    for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens; ++tokenIdx)
    {
        // reset full shared memory field to 0
        smemExpertTokenCountFull[tokenIdx][threadIdx.x] = 0;
    }
    __syncwarp();

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }
#endif

    if (params.mPtrScores != nullptr)
    {
        // if we use `mPtrScores` as input, we need to perform the top-1 reduction
        // for each token, we load the scores then use `reduceTopK` for this.
        // each thread works on 4 experts, so a local reduction is done before
        for (int tokenIdx = 0; tokenIdx < params.mNumTokens; ++tokenIdx)
        {
            auto scoreOffset = tokenIdx * params.mNumExperts;
            int32_t warpMaxExpertIdx[MaxNumTopExperts];
            InputT warpMaxScore[MaxNumTopExperts];

            // Use routingTopKExperts function instead of inline logic
            routingTopKExperts<InputT, ExpertsPerThread>(
                warp, warpMaxScore, warpMaxExpertIdx, threadIdx.x, params.mNumExperts, params.mPtrScores + scoreOffset);

            if (cute::elect_one_sync())
            {
                // one thread updates the count linking token to chosen expert
                auto expertTokenCount = 0;
                setBits</* IsZero= */ true>(expertTokenCount, 1, warpMaxExpertIdx[0] % ExpertsPerThread);
                smemExpertTokenCountFull[tokenIdx][warpMaxExpertIdx[0] / ExpertsPerThread] = expertTokenCount;
                // we also compute the final score here and write it out if required
                auto finalScore = OutputT{sigmoid_accurate(float{warpMaxScore[0]})};
                if (params.mPtrExpertWeights != nullptr)
                {
                    params.mPtrExpertWeights[tokenIdx] = finalScore;
                }
            }
        }
    }
    else
    {
        // if we do not have `mPtrScores` as input, we expect that `mPtrExpertWeights`
        // contains the top-1 packed score and index already.
        // Each thread represents a token here, and we extract the relevant score
        // The assumption is that the #tokens is limited by warp-size
        static_assert(WarpKernelMaxNumTokens <= WarpSize);
        TypePacked scoreIdx = threadIdx.x < params.mNumTokens ? params.mPtrExpertIdx[threadIdx.x] : TypePacked{};
        int32_t expertTokenCount = 0;
        setBits</* IsZero= */ true>(expertTokenCount, 1, scoreIdx.idx % ExpertsPerThread);
        if (threadIdx.x < params.mNumTokens)
        {
            smemExpertTokenCountFull[threadIdx.x][scoreIdx.idx / ExpertsPerThread] = expertTokenCount;
        }
        // we also compute the final score here and write it out if required
        auto finalScore = OutputT{sigmoid_accurate(float{scoreIdx.score})};
        if (params.mPtrExpertWeights != nullptr && threadIdx.x < params.mNumTokens)
        {
            params.mPtrExpertWeights[threadIdx.x] = finalScore;
        }
    }

    // make the full table available to all threads
    __syncwarp();

    // at this point, each thread keeps a count of its 4 assigned experts in
    // `expertCount`, as well as the offsets for all tokens w.r.t. these 4 experts
    // in `expertOffset`.
    int32_t expertCount = 0;
    int32_t expertOffset[WarpKernelMaxNumTokens + 1];
#pragma unroll
    for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens + 1; ++tokenIdx)
    {
        if (tokenIdx > params.mNumTokens)
            break;
        // simple reduction for `expertCount`, and scan for `expertOffset`
        auto expertTokenCount = tokenIdx < params.mNumTokens ? smemExpertTokenCountFull[tokenIdx][threadIdx.x] : 0;
        expertOffset[tokenIdx] = expertCount;
        expertCount += expertTokenCount;
    }

    // at this point, we are ready for the scan across all experts to get the
    // thread-wise offsets across experts
    // first, we need to reduce across our 4 experts into `numCta`
    int32_t numCta = 0;
#pragma unroll
    for (int ii = 0; ii < ExpertsPerThread; ++ii)
    {
        auto count = getBits(expertCount, ii);
        numCta += divUpLog2<int32_t>(count, params.mPaddingLog2);
    }
    // second, we perform the exclusive sum across the warp
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    // finally, we perform a scan across our local experts, starting with the
    // warp-wide scan result (`ctaOffset`)
    auto ctaOffsetExp = ctaOffset;
#pragma unroll
    for (int ii = 0; ii < ExpertsPerThread; ++ii)
    {
        auto count = getBits(expertCount, ii);
        auto finalNumCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
        auto expertIdx = threadIdx.x * ExpertsPerThread + ii;
        // during the scan for expert offsets, we can already write out
        // both `mPtrCtaIdxXyToBatchIdx` and `mPtrCtaIdxXyToMnLimit`
        for (int cta = 0; cta < finalNumCta; ++cta)
        {
            params.mPtrCtaIdxXyToBatchIdx[ctaOffsetExp + cta] = expertIdx;
            params.mPtrCtaIdxXyToMnLimit[ctaOffsetExp + cta]
                = min(mulLog2<int32_t>(ctaOffsetExp + cta + 1, params.mPaddingLog2),
                    mulLog2<int32_t>(ctaOffsetExp, params.mPaddingLog2) + count);
        }
        ctaOffsetExp += finalNumCta;
    }

    // at this point, we can write out padded count from the warp-aggregate
    if (cute::elect_one_sync())
    {
        const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
#if !defined(PDL_PROFILE) || PDL_PROFILE == 0
    // we can trigger the next kernel at this point
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
#endif
#endif

    // at this point, all values for offsets are ready, except the final offsets
    // within the padded index (`permutedIdx`)
    // for this, we perform a scan similar to the one directly after the warp-scan:
    // here, we keep the local offset for each of the thread's experts in a field
    // of registers
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    int32_t finalExpertOffset[ExpertsPerThread];
    finalExpertOffset[0] = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
#pragma unroll
    for (int ii = 1; ii < ExpertsPerThread; ++ii)
    {
        finalExpertOffset[ii]
            = finalExpertOffset[ii - 1] + divUpMulLog2<int32_t>(getBits(expertCount, ii - 1), params.mPaddingLog2);
    }

#pragma unroll
    for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens; ++tokenIdx)
    {
        // at this point, we can calculate the final index:
        // we simply loop over all tokens, and all experts assigned to this thread.
        // For each pair, we determine whether that token was routed to that expert
        // based on whether the offset for that token changed.
        // we can then easily compute the final `expertIdx` and `permutedIdx` relative
        // to this token and expert, and write them out.
        if (tokenIdx >= params.mNumTokens)
            break;

#pragma unroll
        for (int ii = 0; ii < ExpertsPerThread; ++ii)
        {
            // determine whether the offset for this expert and token changes
            auto localOffsetToken = getBits(expertOffset[tokenIdx], ii);
            auto isTokenRouted = getBits(expertOffset[tokenIdx + 1], ii) > localOffsetToken;
            // the expert index of this expert
            auto expertIdx = threadIdx.x * ExpertsPerThread + ii;
            auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            // the permuted index: we add the local offset relative to this expert and token
            // to the global offset from the scan for this expert
            auto permutedIdx = isLocalExpert ? finalExpertOffset[ii] + localOffsetToken : int32_t{-1};
            // write out `mPtrExpandedIdxToPermutedIdx` if required
            if (params.mPtrExpandedIdxToPermutedIdx != nullptr && isTokenRouted)
            {
                params.mPtrExpandedIdxToPermutedIdx[tokenIdx] = permutedIdx;
            }
            // write out `mPtrPermutedIdxToTokenIdx` if required
            if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert && isTokenRouted)
            {
                params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
            }
        }
    }
}
////////////////////////////////////////////////////////////////////////////////////////////////////
template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params)
{
    // number of tokens/expanded idx is bounded by total number of warps
    using OutputT = typename KernelParams::OutputT;
    using InputT = typename KernelParams::InputT;
    using TypePacked = PackedScoreIdx<OutputT>;
    __shared__ TypePacked __attribute((aligned(128))) smemPackedScoreIdx[NumWarps];

    uint32_t const clusterBlockRank = blockIdx.x;
    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
    int32_t const laneIdx = cutlass::arch::LaneId();

    // TODO(mjoux): expand to more tokens (possibly)
    auto warpTokenIdx = clusterBlockRank * NumWarps + warpIdx;
    auto scoreOffset = warpTokenIdx * params.mNumExperts;
    bool validToken = warpTokenIdx < params.mNumTokens;
    InputT minScore = InputT{-INFINITY};

    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    if (params.mPtrScores != nullptr)
    {
        // in this case, each warp represents a token
        // we then exchange all token max scores, s.t. afterwards, each thread
        // represents a token
        InputT warpMaxScore[MaxNumTopExperts];
        int32_t warpMaxExpertIdx[MaxNumTopExperts];

        if (validToken)
        {
            routingTopKExperts<InputT, MaxNumExperts / WarpSize>(
                warp, warpMaxScore, warpMaxExpertIdx, laneIdx, params.mNumExperts, params.mPtrScores + scoreOffset);
            if (cute::elect_one_sync())
            {
                auto finalScore = OutputT{sigmoid_accurate(float{warpMaxScore[0]})};
                TypePacked packedScore{finalScore, static_cast<int16_t>(warpMaxExpertIdx[0])};
                smemPackedScoreIdx[warpIdx] = packedScore;
            }
        }
        // make packed scores available to all threads in cluster
        __cluster_barrier_arrive();
        __cluster_barrier_wait();
    }

    routingPermutation<KernelParams, OutputT, NumThreads, NumWarps, MaxNumTopExperts,
        /*LoadExpertIdxFromGlobal=*/false>(params, smemPackedScoreIdx, warpIdx, clusterBlockRank);
}
#else
__global__ void routingIndicesClusterKernel(KernelParams params)
{
    assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

// this kernel is needed in case we have scores as input for the histogram kernel
template <typename KernelParams>
__global__ void __launch_bounds__(NumThreadsHist) routingIndicesHistogramScoresKernel(KernelParams params)
{
    using OutputT = typename KernelParams::OutputT;
    using InputT = typename KernelParams::InputT;
    using TypePacked = PackedScoreIdx<OutputT>;
    static constexpr int VecSize = MaxNumExperts / WarpSize;
    //  we assume that #experts is a multiple of 4, so VecSize must be 4.
    static_assert(VecSize == 4);

    int32_t const laneIdx = cutlass::arch::LaneId();
    int32_t const warpIdx = threadIdx.x / WarpSize;
    int32_t const globalWarpIdx = blockIdx.x * NumWarpsHist + warpIdx;
    int32_t const globalWarpStride = gridDim.x * NumWarpsHist;
    InputT minScore = InputT{-INFINITY};
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

    // initialize the mPtrExpertCounts
    int32_t expertCountsNum = 2 * params.mNumExperts;
    int32_t globalThreadIdx = blockIdx.x * NumThreads + threadIdx.x;
    int32_t globalThreadStride = gridDim.x * NumThreads;
    initArr(globalThreadIdx, expertCountsNum, globalThreadStride, params.mPtrExpertCounts, 0);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    // Wait on primary grid and trigger secondary kernel.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
        cudaTriggerProgrammaticLaunchCompletion();
    }
#endif

    // in this case, each warp represents a token, and we use a grid-stride loop
    // over all warps/tokens
    for (int tokenIdx = globalWarpIdx; tokenIdx < params.mNumTokens; tokenIdx += globalWarpStride)
    {
        auto scoreOffset = tokenIdx * params.mNumExperts;
        int32_t warpMaxExpertIdx[MaxNumTopExperts];
        InputT warpMaxScore[MaxNumTopExperts];

        routingTopKExperts<InputT, MaxNumExperts / WarpSize>(
            warp, warpMaxScore, warpMaxExpertIdx, laneIdx, params.mNumExperts, params.mPtrScores + scoreOffset);

        if (cute::elect_one_sync())
        {
            auto finalScore = OutputT{sigmoid_accurate(float{warpMaxScore[0]})};
            TypePacked packedScore{finalScore, static_cast<int16_t>(warpMaxExpertIdx[0])};
            params.mPtrExpertIdx[tokenIdx] = packedScore;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    TLLM_CHECK_WITH_INFO(data.mPtrExpertIdx != nullptr || data.mPtrScores != nullptr,
        "Routing kernel requires at least one input parameter");
    TLLM_CHECK_WITH_INFO(data.mPtrPermutedIdxSize != nullptr && data.mPtrCtaIdxXyToBatchIdx != nullptr
            && data.mPtrCtaIdxXyToMnLimit != nullptr && data.mPtrNumNonExitingCtas != nullptr,
        "Llama4 routing kernel expects permuted idx and grouped Gemm launch config buffers");
    TLLM_CHECK_WITH_INFO(data.mTopK <= MaxNumTopExperts, "Routing kernel expects topK experts <= %d, got %d",
        MaxNumTopExperts, data.mTopK);
    TLLM_CHECK_WITH_INFO(data.mNumExperts <= MaxNumExperts,
        "Routing kernel expects #experts %d to be at most max #experts %d", data.mNumExperts, MaxNumExperts);
    static_assert(MaxNumExperts <= NumThreads, "#experts must be bounded by #threads");
    static_assert(MaxNumExperts <= NumThreadsHist, "#experts must be bounded by #threads");
    TLLM_CHECK_WITH_INFO(
        data.mNumExperts % 4 == 0, "Routing kernel expects #experts %d to be a multiple of 4.", data.mNumExperts);
    TLLM_CHECK_WITH_INFO(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got %d", data.mPaddingLog2);

    bool const useSingleWarp = (data.mPtrScores == nullptr && data.mNumTokens <= WarpKernelMaxNumTokens)
        || data.mNumTokens < WarpKernelMaxNumTokens;
    bool const useSingleCluster
        = data.mNumTokens <= (data.mPtrScores != nullptr ? MaxNumTokensSingleClusterScores : MaxNumTokensSingleCluster);
    if (!useSingleCluster)
    {
        TLLM_CHECK_WITH_INFO(
            data.mPtrExpertIdx != nullptr, "When #tokens is large, `mPtrExpertIdx` is a required input.");
        TLLM_CHECK_WITH_INFO(
            data.mPtrExpertCounts != nullptr, "When #tokens is large, `mPtrExpertCounts` is a required input.");
    }

    if (useSingleWarp)
    {
        LAUNCH_ROUTING(data,
            /*coopLaunch=*/false, routingIndicesWarpKernel, 1, WarpSize,
            /*smemSize=*/0, // No dynamic smem
            stream);
    }
    else if (useSingleCluster)
    {
        LAUNCH_ROUTING(data,
            /*coopLaunch=*/false, routingIndicesClusterKernel, NumBlocksPerCluster, NumThreads,
            /*smemSize=*/0, // No dynamic smem
            stream);
    }
    else
    {
        const uint32_t expandedIdxSize = data.mNumTokens * data.mTopK;

        const uint32_t histogramEltsPerBlock = 8 * NumThreadsHist;
        const uint32_t offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreadsHist;

        // Limit grid size (all kernels use a grid-stride loop).
        const uint32_t maxNumBlocks = 1024;

        int const numBlocksHistogram
            = std::min((expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
        int const numBlocksOffsets
            = std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

        if (data.mPtrScores != nullptr)
        {
            LAUNCH_ROUTING(data,
                /*coopLaunch=*/false, routingIndicesHistogramScoresKernel, maxNumBlocks, NumThreadsHist,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
        else
        {
            // Reset the global histograms.
            TLLM_CUDA_CHECK(hipMemsetAsync(data.mPtrExpertCounts, 0,
                static_cast<size_t>(2 * NumThreads) * sizeof(int32_t), (hipStream_t) stream));
        }
        LAUNCH_ROUTING(data,
            /*coopLaunch=*/false, routingIndicesHistogramKernel, numBlocksHistogram, NumThreadsHist,
            /*smemSize=*/0, // No dynamic smem
            stream);
        LAUNCH_ROUTING(data,
            /*coopLaunch=*/false, routingIndicesOffsetsKernel, numBlocksOffsets, NumThreadsHist,
            /*smemSize=*/0, // No dynamic smem
            stream);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace routingLlama4
} // namespace moe::dev::routing
