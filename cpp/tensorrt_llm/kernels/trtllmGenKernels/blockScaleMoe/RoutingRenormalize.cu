#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "RoutingKernel.cuh"

namespace moe::dev::routing
{
namespace routingRenormalize
{
////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 1024;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int MaxNumTopExperts = 8;
static constexpr int MaxNumExperts = 128;
static constexpr int MaxNumTokensSingleCluster = NumBlocksPerCluster * NumThreads;
static constexpr int MaxNumTokensSingleClusterScores = NumBlocksPerCluster * NumWarps;

template <typename DataType, typename InputType, int VecSize, bool DoSoftmaxBeforeTopK>
__forceinline__ __device__ void routingTopKExperts(cg::thread_block_tile<WarpSize> const& warp,
    DataType (&score)[VecSize], int32_t (&idx)[VecSize], DataType (&warpTopKScore)[MaxNumTopExperts],
    int32_t (&warpTopKExpertIdx)[MaxNumTopExperts], int32_t const laneIdx, int32_t const numExperts, int32_t topK,
    InputType const* ptrScores, bool const normTopkProb)
{
    DataType minScore = DataType{-INFINITY};

    for (int i = 0; i < VecSize; i++)
    {
        auto expertIdx = i * WarpSize + laneIdx;
        auto newScore = expertIdx < numExperts ? static_cast<DataType>(ptrScores[expertIdx]) : minScore;
        score[i] = newScore;
        idx[i] = expertIdx;
    }
    if constexpr (DoSoftmaxBeforeTopK)
    {
        calcSoftmax(warp, score);
    }

    // Get the top-k scores and their corresponding expert indices
    topk::reduceTopK(warp, warpTopKScore, warpTopKExpertIdx, score, idx, minScore, topK);

    // Normalize the scores
    if constexpr (DoSoftmaxBeforeTopK)
    {
        float sum = float{1.f};
        if (normTopkProb)
        {
            sum = static_cast<float>(laneIdx < topK ? warpTopKScore[laneIdx] : 0);
            sum = cg::reduce(warp, sum, cg::plus<float>());
        }
        if (laneIdx < topK)
        {
            warpTopKScore[laneIdx] = warpTopKScore[laneIdx] / sum;
        }
    }
    else
    {
        auto softmaxScore = calcSoftmax(warp, laneIdx < topK ? warpTopKScore[laneIdx] : minScore, laneIdx, topK);
        if (laneIdx < topK)
        {
            warpTopKScore[laneIdx] = softmaxScore;
        }
    }
}

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params)
{
    // number of tokens/expanded idx is bounded by total number of warps
    using OutputT = typename KernelParams::OutputT;
    using InputT = typename KernelParams::InputT;

    using BaseType = std::conditional_t<KernelParams::DoSoftmaxBeforeTopK, float, InputT>;
    using TypePacked = PackedScoreIdx<BaseType>;

    static constexpr int VecSize = MaxNumExperts / WarpSize;
    // we assume that #experts is a multiple of 4, so VecSize must be 4.
    static_assert(VecSize == 4);

    __shared__ TypePacked __attribute((aligned(128))) smemPackedScoreIdx[NumWarps * MaxNumTopExperts];

    uint32_t const clusterBlockRank = blockIdx.x;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
    int32_t const laneIdx = cutlass::arch::LaneId();

    auto warpTokenIdx = clusterBlockRank * NumWarps + warpIdx;
    auto scoreOffset = warpTokenIdx * params.mNumExperts;
    bool validToken = warpTokenIdx < params.mNumTokens;

    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    if (params.mPtrScores != nullptr)
    {
        // in this case, each warp represents a token
        BaseType score[VecSize];
        int32_t idx[VecSize];

        BaseType warpTopKScore[MaxNumTopExperts];
        int32_t warpTopKExpertIdx[MaxNumTopExperts];

        BaseType minScore = BaseType{-INFINITY};
        if (validToken)
        {
            routingTopKExperts<BaseType, InputT, VecSize, KernelParams::DoSoftmaxBeforeTopK>(warp, score, idx,
                warpTopKScore, warpTopKExpertIdx, laneIdx, params.mNumExperts, params.mTopK,
                params.mPtrScores + scoreOffset, params.mNormTopkProb);

            if (laneIdx < params.mTopK)
            {
                smemPackedScoreIdx[warpIdx * params.mTopK + laneIdx]
                    = TypePacked{warpTopKScore[laneIdx], static_cast<int16_t>(warpTopKExpertIdx[laneIdx])};
            }
        } // end if (validToken)

        // make packed scores available to all threads in cluster
        __cluster_barrier_arrive();
        __cluster_barrier_wait();
    }

    routingPermutation<KernelParams, BaseType, NumThreads, NumWarps, MaxNumTopExperts,
        /*LoadExpertIdxFromGlobal=*/false>(params, smemPackedScoreIdx, warpIdx, clusterBlockRank);
}
#else
__global__ void __launch_bounds__(NumThreads) routingIndicesClusterKernel(KernelParams /* params */)
{
    assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
////////////////////////////////////////////////////////////////////////////////////////////////////

// this kernel is needed in case we have scores as input for the histogram kernel
template <typename KernelParams>
__global__ void __launch_bounds__(NumThreadsHist) routingIndicesHistogramScoresKernel(KernelParams params)
{
    using OutputT = typename KernelParams::OutputT;
    using InputT = typename KernelParams::InputT;
    using BaseType = std::conditional_t<KernelParams::DoSoftmaxBeforeTopK, float, InputT>;

    static constexpr int VecSize = MaxNumExperts / WarpSize;
    // we assume that #experts is a multiple of 4, so VecSize must be 4.
    static_assert(VecSize == 4);

    int32_t const laneIdx = cutlass::arch::LaneId();
    int32_t const warpIdx = threadIdx.x / WarpSize;
    int32_t const globalWarpIdx = blockIdx.x * NumWarpsHist + warpIdx;
    int32_t const globalWarpStride = gridDim.x * NumWarpsHist;
    BaseType minScore = BaseType{-INFINITY};
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    // Wait on primary grid.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))

    // initialize the mPtrExpertCounts
    int32_t expertCountsNum = 2 * params.mNumExperts;
    int32_t globalThreadIdx = blockIdx.x * NumThreads + threadIdx.x;
    int32_t globalThreadStride = gridDim.x * NumThreads;
    initArr(globalThreadIdx, expertCountsNum, globalThreadStride, params.mPtrExpertCounts, 0);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    // Trigger secondary kernel.
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))

    // in this case, each warp represents a token, and we use a grid-stride loop
    // over all warps/tokens
    BaseType allScores[VecSize];
    int32_t allExpertIdx[VecSize];
    BaseType warpTopKScore[MaxNumTopExperts];
    int32_t warpTopKExpertIdx[MaxNumTopExperts];
    for (int tokenIdx = globalWarpIdx; tokenIdx < params.mNumTokens; tokenIdx += globalWarpStride)
    {
        auto scoreOffset = tokenIdx * params.mNumExperts;

        routingTopKExperts<BaseType, InputT, VecSize, KernelParams::DoSoftmaxBeforeTopK>(warp, allScores, allExpertIdx,
            warpTopKScore, warpTopKExpertIdx, laneIdx, params.mNumExperts, params.mTopK,
            params.mPtrScores + scoreOffset, params.mNormTopkProb);

        if (laneIdx < params.mTopK)
        {
            PackedScoreIdx<OutputT> packedScore{
                static_cast<OutputT>(warpTopKScore[laneIdx]), static_cast<int16_t>(warpTopKExpertIdx[laneIdx])};
            params.mPtrExpertIdx[tokenIdx * params.mTopK + laneIdx] = packedScore;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    TLLM_CHECK_WITH_INFO(data.mPtrExpertIdx != nullptr || data.mPtrScores != nullptr,
        "Routing kernel requires at least one input parameter");
    TLLM_CHECK_WITH_INFO(data.mPtrPermutedIdxSize != nullptr && data.mPtrCtaIdxXyToBatchIdx != nullptr
            && data.mPtrCtaIdxXyToMnLimit != nullptr && data.mPtrNumNonExitingCtas != nullptr,
        "Llama4 routing kernel expects permuted idx and grouped Gemm launch config buffers");
    TLLM_CHECK_WITH_INFO(data.mTopK <= MaxNumTopExperts, "Routing kernel expects topK experts <= %d, got %d",
        MaxNumTopExperts, data.mTopK);
    TLLM_CHECK_WITH_INFO(data.mNumExperts <= MaxNumExperts,
        "Routing kernel expects #experts %d to be at most max #experts %d", data.mNumExperts, MaxNumExperts);
    static_assert(MaxNumExperts <= NumThreads, "#experts must be bounded by #threads");
    static_assert(MaxNumExperts <= NumThreadsHist, "#experts must be bounded by #threads");
    TLLM_CHECK_WITH_INFO(
        data.mNumExperts % 4 == 0, "Routing kernel expects #experts %d to be a multiple of 4.", data.mNumExperts);
    TLLM_CHECK_WITH_INFO(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got %d", data.mPaddingLog2);

    bool const useSingleCluster
        = data.mNumTokens <= (data.mPtrScores != nullptr ? MaxNumTokensSingleClusterScores : MaxNumTokensSingleCluster);

    if (!useSingleCluster)
    {
        TLLM_CHECK_WITH_INFO(
            data.mPtrExpertIdx != nullptr, "When #tokens is large, `mPtrExpertIdx` is a required input.");
        TLLM_CHECK_WITH_INFO(
            data.mPtrExpertCounts != nullptr, "When #tokens is large, `mPtrExpertCounts` is a required input.");
    }

    if (useSingleCluster)
    {
        LAUNCH_ROUTING_WITH_EXTRA_FLAG(data, false, routingIndicesClusterKernel, NumBlocksPerCluster, NumThreads,
            /*smemSize=*/0, // No dynamic smem
            stream, data.mDoSoftmaxBeforeTopK, /*forceFloatInput=*/false);
    }
    else
    {
        uint32_t const expandedIdxSize = data.mNumTokens * data.mTopK;

        uint32_t const histogramEltsPerBlock = 8 * NumThreadsHist;
        uint32_t const offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreadsHist;

        // Limit grid size (all kernels use a grid-stride loop).
        uint32_t const maxNumBlocks = 1024;

        int const numBlocksHistogram
            = std::min((expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
        int const numBlocksOffsets
            = std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

        if (data.mPtrScores != nullptr)
        {
            LAUNCH_ROUTING_WITH_EXTRA_FLAG(data, false, routingIndicesHistogramScoresKernel, maxNumBlocks,
                NumThreadsHist,
                /*smemSize=*/0, // No dynamic smem
                stream, data.mDoSoftmaxBeforeTopK, /*forceFloatInput=*/false);
        }
        else
        {
            // Reset the global histograms.
            TLLM_CUDA_CHECK(hipMemsetAsync(data.mPtrExpertCounts, 0,
                static_cast<size_t>(2 * NumThreads) * sizeof(int32_t), (hipStream_t) stream));
        }
        LAUNCH_ROUTING_WITH_EXTRA_FLAG(data, false, routingIndicesHistogramKernel, numBlocksHistogram, NumThreadsHist,
            /*smemSize=*/0, // No dynamic smem
            stream, data.mDoSoftmaxBeforeTopK, /*forceFloatInput=*/false);
        LAUNCH_ROUTING_WITH_EXTRA_FLAG(data, false, routingIndicesOffsetsKernel, numBlocksOffsets, NumThreadsHist,
            /*smemSize=*/0, // No dynamic smem
            stream, data.mDoSoftmaxBeforeTopK, /*forceFloatInput=*/false);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace routingRenormalize
} // namespace moe::dev::routing
