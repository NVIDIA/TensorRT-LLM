#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "DevKernel.h"
#include "RoutingKernel.h"

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>

#include <cute/arch/cluster_sm90.hpp>
#include <cutlass/arch/arch.h>

#include <type_traits>

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace moe::dev
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace routing
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace tg = batchedGemm::trtllm::gen;
namespace cg = cooperative_groups;

////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 256;
static constexpr int NumBlocksPerCluster = 8;
static constexpr int WarpSize = 32;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int NumTopGroupScores = 2;
static constexpr int MaxNumTopExperts = 8;
static constexpr int MaxNumTopGroups = 4;

// Performance tuning knob.
static constexpr int NumEltsPerOffsetTilePerThread = 8;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 1000 && defined(__CUDA_ARCH_FEAT_SM100_ALL))
#define TLLM_GEN_ENABLE_FAST_REDUX
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename TypeExpW_>
struct TopKRedType
{
    using TypeExpW = TypeExpW_;
    static_assert(std::is_same_v<TypeExpW, float> || std::is_same_v<TypeExpW, cutlass::bfloat16_t>,
        "Top K reduction only implemented for float and Bf16");
    using TypeCmp = std::conditional_t<sizeof(TypeExpW) >= 4, double, float>;
    static constexpr int64_t Mask64 = 0x000000000000FFFF;
    static constexpr int32_t Mask32 = 0x0000FFFF;

    TypeCmp compVal;

    static __host__ __device__ inline TypeCmp makeCmpVal(TypeExpW val, int32_t idx = 0)
    {
        auto cmpVal = TypeCmp{val};
        TypeCmp cmpValWithIdx;
        if constexpr (sizeof(TypeExpW) >= 4)
        {
            auto cmpValIdx64 = reinterpret_cast<int64_t&>(cmpVal) | (Mask64& int64_t{idx});
            cmpValWithIdx = reinterpret_cast<TypeCmp&>(cmpValIdx64);
        }
        else
        {
            auto cmpValIdx32 = reinterpret_cast<int32_t&>(cmpVal) | (Mask32 & idx);
            cmpValWithIdx = reinterpret_cast<TypeCmp&>(cmpValIdx32);
        }
        return cmpValWithIdx;
    }

    static __host__ __device__ inline void unpack(TypeExpW& val, int32_t& idx, TypeCmp cmp)
    {
        if constexpr (sizeof(TypeExpW) >= 4)
        {
            idx = static_cast<int32_t>(reinterpret_cast<int64_t&>(cmp) & Mask64);
            auto val64 = reinterpret_cast<int64_t&>(cmp) & ~Mask64;
            val = static_cast<float>(reinterpret_cast<double&>(val64));
        }
        else
        {
            idx = reinterpret_cast<int32_t&>(cmp) & Mask32;
            auto val32 = reinterpret_cast<int32_t&>(cmp) >> 16;
            val = TypeExpW::bitcast(reinterpret_cast<uint16_t&>(val32));
        }
    }

    __host__ __device__ TopKRedType() = default;

    __host__ __device__ TopKRedType(TypeExpW val, int32_t idx)
        : compVal(makeCmpVal(val, idx))
    {
    }

    __host__ __device__ operator TypeCmp() const noexcept
    {
        return compVal;
    }

    __device__ inline TypeCmp reduce(cg::thread_block_tile<WarpSize> const& warp)
    {
#if defined(TLLM_GEN_ENABLE_FAST_REDUX)
        static constexpr bool UseCg = false;
#else
        static constexpr bool UseCg = true;
#endif
        if constexpr (UseCg || sizeof(TypeExpW) >= 4)
        {
            return cg::reduce(warp, compVal, cg::greater<TypeCmp>{});
        }
        else
        {
            float result;
            asm("redux.sync.max.f32 %0, %1, 0xffffffff;\n" : "=f"(result) : "f"(compVal));
            return result;
        }
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

static __device__ inline float sigmoid_accurate(float x)
{
    return 0.5f * tanhf(0.5f * x) + 0.5f;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K_, bool Enable_>
struct TopKIdx
{
    // by default, empty
};

template <int K_>
struct TopKIdx<K_, true>
{
    static constexpr int K = K_;
    int32_t val[K];
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K, typename Type>
__device__ void reduceTopK(cg::thread_block_tile<WarpSize> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type value, int32_t idx, Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WarpSize, "Top K must have K < WarpSize");
    using RedType = TopKRedType<Type>;
    RedType topK{value, idx};
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        topK = kk > 0 && packedMax == topK.compVal ? RedType{minValue, idx} : topK;
        // get the next largest value
        packedMax = topK.reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

#define TOPK_SWAP(I, J)                                                                                                \
    {                                                                                                                  \
        auto pairMin = min(topK[I].compVal, topK[J].compVal);                                                          \
        auto pairMax = max(topK[I].compVal, topK[J].compVal);                                                          \
        topK[I].compVal = pairMax;                                                                                     \
        topK[J].compVal = pairMin;                                                                                     \
    }

template <int K, typename Type, int N, bool IsSorted = false>
__device__ void reduceTopK(cg::thread_block_tile<WarpSize> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type (&value)[N], int32_t (&idx)[N], Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WarpSize, "Top K must have K < WarpSize");
    static_assert(N > 0, "Top K must have N > 1");
    static_assert(N <= K, "Top K must have N < K");
    using RedType = TopKRedType<Type>;
    RedType topK[N];
#pragma unroll
    for (int nn = 0; nn < N; ++nn)
        topK[nn] = RedType{value[nn], idx[nn]};
    if constexpr (!IsSorted)
    {
        static_assert(N <= 4, "Unsorted topK expects N <= 4");
        TOPK_SWAP(0, 2);
        TOPK_SWAP(1, 3);

        TOPK_SWAP(0, 1);
        TOPK_SWAP(2, 3);

        TOPK_SWAP(1, 2);
    }
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        bool update = kk > 0 && packedMax == topK[0].compVal;
#pragma unroll
        for (int nn = 0; nn < N; ++nn)
        {
            topK[nn] = update && nn == N - 1 ? RedType{minValue, idx[nn]} : update ? topK[nn + 1] : topK[nn];
        }
        // get the next largest value
        packedMax = topK[0].reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

#undef TOPK_SWAP

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T mulLog2(T a, T bLog2)
{
    return a << bLog2;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T divUpLog2(T a, T bLog2)
{
    return ((a + (1 << bLog2) - 1) >> bLog2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T divUpMulLog2(T a, T bLog2)
{
    return mulLog2<T>(divUpLog2<T>(a, bLog2), bLog2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void routingMainKernel(KernelParams params)
{
    // declare types required for reductions
    using TypeExpW = typename KernelParams::TypeExpW;

    // declare shared memory structure
    // number of experts is bounded by number of threads
    __shared__ float __attribute((aligned(128))) smemScoreSigmoid[NumThreads];
    __shared__ float __attribute((aligned(128))) smemScoreBias[NumThreads];
    // number of expert groups is bounded by number of warps
    __shared__ float __attribute((aligned(128))) smemGroupScores[NumWarps];

    // needed for warp reduce
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);
    // for the final reduction of weight norm, only some lanes need to participate
    int32_t laneIdx = threadIdx.x % WarpSize;
    int32_t warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
    // warps outside the range of expert groups do not participate
    if constexpr (KernelParams::UseGroups)
    {
        if (warpIdx >= params.mNumExpertGroups)
        {
            return;
        }
    }

    // note that for invalid scores, we simply use a negative value:
    // they work well even with the compacted format used in topK, and
    // sigmoid / bias activated scores cannot be negative
    static constexpr float invalidScoreFloat = -1.F;
    const TypeExpW invalidScore = TypeExpW{invalidScoreFloat};

    // load bias already; each warp represents one expert group
    auto threadExpert = threadIdx.x;
    bool expertSelected = threadExpert < params.mNumExperts;
    if constexpr (KernelParams::UseGroups)
    {
        threadExpert = warpIdx * params.mNumExpertsPerGroup + laneIdx;
        expertSelected = laneIdx < params.mNumExpertsPerGroup;
    }
    auto scoreIdx = int64_t{blockIdx.x} * int64_t{params.mNumExperts} + threadExpert;
    auto biasVal = expertSelected ? params.mPtrRoutingBias[threadExpert] : invalidScore;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    // trigger the secondary kernel when using PDL, then wait on primary
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
        cudaGridDependencySynchronize();
    }
#endif

    // get our assigned thread score; each warp represents one expert group
    float score = expertSelected ? params.mPtrScores[scoreIdx] : invalidScoreFloat;
    // get the sigmoid score
    // note that for invalid values, we simply use a negative value:
    // sigmoig scores are always strictly positive
    auto scoreSigmoid = sigmoid_accurate(score);
    // write the sigmoid score to shared for later use
    if (expertSelected)
    {
        smemScoreSigmoid[threadExpert] = scoreSigmoid;
    }
    // get the score with bias
    // note that with invalid values, because sigmoid is < 1 and bias is -1,
    // we must get a negative value, which is smaller than any valid value
    auto scoreBias = float{scoreSigmoid + float{biasVal}};
    if (expertSelected)
    {
        smemScoreBias[threadExpert] = scoreBias;
    }

    // registers for top group score reduction
    float topExpGroupScores[NumTopGroupScores];
    [[maybe_unused]] int32_t topExpGroupIdx[NumTopGroupScores];
    float topGroups[MaxNumTopGroups]; // bound of params.mNumLimitedGroups
    int32_t topGroupIdx[MaxNumTopGroups];
    float expertScoreGroup[MaxNumTopGroups];
    int32_t expertIdxGroup[MaxNumTopGroups];
    float topScores[MaxNumTopExperts]; // bound of params.mTopK
    int32_t topExperts[MaxNumTopExperts];

    if constexpr (KernelParams::UseGroups)
    {
        reduceTopK(warp, topExpGroupScores, topExpGroupIdx, scoreBias, threadExpert,
            /* minValue */ invalidScoreFloat);

        // get the final group score and write it to shared
        if (cute::elect_one_sync())
        {
            auto groupScore = topExpGroupScores[0] + topExpGroupScores[1];
            smemGroupScores[warpIdx] = groupScore;
        }
    }

    // make group scores available to all warps
    __syncthreads();

    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    if (warpIdx == 0)
    {
        // a single warp performs the selection of top groups, and goes on to select the final experts
        if constexpr (KernelParams::UseGroups)
        {
            float groupScore = laneIdx < params.mNumExpertGroups ? smemGroupScores[laneIdx] : invalidScoreFloat;

            reduceTopK(warp, topGroups, topGroupIdx, groupScore, laneIdx,
                /* minValue */ invalidScoreFloat);

            // final expert selection: get relevant indexes and scores from shared

#pragma unroll
            for (int ii = 0; ii < MaxNumTopGroups; ++ii)
            { // bound of params.mNumLimitedGroups
                auto groupIdx = topGroupIdx[ii];
                expertIdxGroup[ii] = groupIdx * params.mNumExpertsPerGroup + laneIdx;
                // note: expertSelected implies laneIdx < params.mNumExpertsPerGroup.
                // we have params.mNumExpertsPerGroup == params.mNumExperts / params.mNumExpertGroups,
                // thus groupIdx <= params.mNumExpertGroups - 1 =>
                // groupIdx * params.mNumExpertsPerGroup <= params.mNumExperts - params.mNumExpertsPerGroup
                // => expertIdxGroup[ii] < params.mNumExperts <= NumThreads,
                // so the access is safe here
                expertScoreGroup[ii] = groupIdx < params.mNumExpertGroups && expertSelected
                    ? smemScoreBias[expertIdxGroup[ii]]
                    : invalidScoreFloat;
            }
        }
        else
        {
            // without groups, each thread just takes `MaxNumTopGroups` experts

#pragma unroll
            for (int ii = 0; ii < MaxNumTopGroups; ++ii)
            {
                auto expertIdx = ii * WarpSize + laneIdx;
                expertIdxGroup[ii] = expertIdx;
                expertScoreGroup[ii] = expertIdx < params.mNumExperts ? smemScoreBias[expertIdx] : invalidScoreFloat;
            }
        }

        reduceTopK(warp, topScores, topExperts, expertScoreGroup, expertIdxGroup,
            /* minValue */ invalidScoreFloat);

        // determine our lane's expert index and write to output
        int32_t expertIdx = 0;
#pragma unroll
        for (int ii = 0; ii < MaxNumTopExperts; ++ii)
        { // bound of params.mTopK
            expertIdx = laneIdx == ii ? topExperts[ii] : expertIdx;
        }
        // determine whether our expert is local to this GPU
        auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;

        // write expert idx out already
        auto idxTopK = blockIdx.x * params.mTopK + laneIdx;
        if (laneIdx < params.mTopK && params.mPtrExpertIdx != nullptr)
        {
            params.mPtrExpertIdx[idxTopK] = expertIdx;
        }
        float scoreNorm = laneIdx < params.mTopK ? smemScoreSigmoid[expertIdx] : 0.F;
        auto redNorm = cg::reduce(warp, scoreNorm, cg::plus<float>{});
        auto finalScore = TypeExpW{scoreNorm * params.mRouteScale / redNorm};
        if (laneIdx < params.mTopK && params.mPtrExpertWeights != nullptr)
        {
            params.mPtrExpertWeights[idxTopK] = finalScore;
        }
        if (laneIdx < params.mTopK && params.mPtrExpertWeightsFull != nullptr && isLocalExpert)
        {
            auto idxWeightsFull = localExpertIdx * gridDim.x + blockIdx.x;
            params.mPtrExpertWeightsFull[idxWeightsFull] = finalScore;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params)
{
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
    // needed for the exclusive sum of token offsets
    using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    // Number of threads in the cluster.
    static constexpr int NumThreadsPerCluster = NumThreads * NumBlocksPerCluster;
    // If the number of tokens is bounded by 16384, then the total number of indexes
    // is bounded by 16384 * TopK.
    // TODO: if we only use this kernel up to 1024 tokens, we could use 1024 here.
    static constexpr int MaxExpandedIdxPerThread
        = (16384 * MaxNumTopExperts + NumThreadsPerCluster - 1) / NumThreadsPerCluster;

    // Initialize cluster.
    int32_t const clusterBlockRank = blockIdx.x;
    int32_t const clusterThreadIdx = NumThreads * clusterBlockRank + threadIdx.x;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

    auto expandedIdxSize = params.mNumTokens * params.mTopK;

    // pre-fill the counts with 0
    smemExpertCount[threadIdx.x] = 0;
    __syncthreads();

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    // each thread keeps has some number of "expanded indexes" assigned to it
    // for each of these, we keep the associated expert and offset within expert in registers
    int32_t expertIndexes[MaxExpandedIdxPerThread];
    int32_t expertOffsets[MaxExpandedIdxPerThread];
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
    // time, and branch between a fast path without bound checks and a slow path with bound checks.
    int constexpr IterStride = 4;
    static_assert(MaxExpandedIdxPerThread % IterStride == 0);

    // Define a lambda to avoid code duplication in both branches.
    auto loopBody = [&](int ii, int expandedIdx)
    {
        int32_t expertIdx = params.mPtrExpertIdx[expandedIdx];
        expertIndexes[ii] = expertIdx;
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + expertIdx, 1) : 0;
    };

#pragma unroll
    for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride)
    {
        // Whether it's safe to do multiple iterations without bound checks.
        bool const takeFastPath = (ii0 + IterStride) * NumThreadsPerCluster <= expandedIdxSize;
        if (takeFastPath)
        {
#pragma unroll
            for (int32_t jj = 0; jj < IterStride; jj++)
            {
                int const ii = ii0 + jj;
                auto expandedIdx = static_cast<int32_t>(clusterThreadIdx) + ii * NumThreadsPerCluster;
                loopBody(ii, expandedIdx);
            }
        }
        else
        {
            bool doBreak = false;
#pragma unroll
            for (int32_t jj = 0; jj < IterStride; jj++)
            {
                int const ii = ii0 + jj;
                auto expandedIdx = static_cast<int32_t>(clusterThreadIdx) + ii * NumThreadsPerCluster;
                if (expandedIdx >= expandedIdxSize)
                {
                    doBreak = true;
                    break;
                }
                loopBody(ii, expandedIdx);
            }
            if (doBreak)
            {
                break;
            }
        }
    }

    // Make local histogram (token counts per expert) available to all threads in the cluster.
    cg::cluster_group::sync();

    //
    // Each thread now represents one expert
    //

    // Get the histogram bin from each rank for this expert.
    int32_t expertCounts[NumBlocksPerCluster];
#pragma unroll
    for (int rank = 0; rank < NumBlocksPerCluster; rank++)
    {
        int32_t const* remoteSmem = cg::cluster_group::map_shared_rank(smemExpertCount, rank);
        expertCounts[rank] = remoteSmem[threadIdx.x];
    }

    // Compute an exclusive prefix sum of the block-local count.
    // Each block only needs the count up to its rank, and the total count.
    int32_t count = 0;
    int32_t blockExpertOffset = 0;
#pragma unroll
    for (int rank = 0; rank < NumBlocksPerCluster; rank++)
    {
        if (rank == clusterBlockRank)
        {
            blockExpertOffset = count;
        }
        count += expertCounts[rank];
    }

    // Arrive: we do not access distributed shared memory after this point.
    __cluster_barrier_arrive();

    // Compute the runtime config for projections
    // Whether or not an expert is local is taken into account when smemExpertCount is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    // Strided loop to share this work between blocks.
    int32_t tokensPerTile = params.mAllToAllRouteAct ? params.mNumTokens : count;
    for (int32_t cta = clusterBlockRank; cta < numCta; cta += NumBlocksPerCluster)
    {
        const int32_t localExpertIdx = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
        params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
        params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta] = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
            mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + tokensPerTile);
    }

    // get the padded offset associated with this expert
    const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
    const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);

    // write out padded count
    if (clusterBlockRank == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync())
    {
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    // write expert offsets to shared
    smemExpertOffset[threadIdx.x] = offset + blockExpertOffset;

    // make expert offsets available to all threads
    __syncthreads();

    // Wait: we cannot exit while other blocks may be accessing the current block's shared memory.
    // Note: I observed a perf benefit to doing this before the final loop so the compiler can
    // implement break with EXIT.
    __cluster_barrier_wait();

    // trigger the secondary kernel when using PDL
    // We can't do it earlier because FC1 depends on the mPtrCtaIdxXyToBatchIdx,
    // mPtrCtaIdxXyToMnLimit, mPtrNumNonExitingCtas and mPtrTotalNumPaddedTokens
    // TODO: this is not sufficient to ensure visibility in the next kernel!

    // TODO: disable PDL for now to avoid race condition in FC1
    if constexpr (KernelParams::UsePdl)
    {
        // cudaTriggerProgrammaticLaunchCompletion();
    }

// each thread has the same "expanded indexes" assigned to it as above
// at this point, we know the final offsets of experts and the offsets within
// experts, which allows writing the final index values
#pragma unroll
    for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ++ii)
    {
        auto expandedIdx = static_cast<int32_t>(clusterThreadIdx) + ii * NumThreadsPerCluster;
        if (expandedIdx >= expandedIdxSize)
        {
            break;
        }
        auto expertIdx = expertIndexes[ii];
        // check whether this expert is local to our GPU at all
        auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        auto tokenIdx = expandedIdx / params.mTopK;
        auto permutedIdx = isLocalExpert ? int32_t{smemExpertOffset[expertIdx]} + expertOffsets[ii] : int32_t{-1};
        if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
        {
            params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
        }
        if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
        {
            params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
        }
    }
}
#else
__global__ void routingIndicesClusterKernel(KernelParams params)
{
    assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __launch_bounds__(NumThreads) routingIndicesCoopKernel(KernelParams params)
{
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
    // needed for the exclusive sum of token offsets
    using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    // 64 elements -> 128+ registers. Above that we may start to see spilling to local memory.
    static constexpr int MaxExpandedIdxPerThread = 64;

    // Initialize grid.
    cg::grid_group grid = cg::this_grid();
    // Note: the following is more efficient than grid.block_index() because we don't use y and z.
    int32_t const gridBlockIdx = blockIdx.x;
    int32_t const gridThreadIdx = NumThreads * gridBlockIdx + threadIdx.x;
    int32_t const numBlocks = gridDim.x;
    int32_t const numThreadsPerGrid = numBlocks * NumThreads;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

    auto expandedIdxSize = params.mNumTokens * params.mTopK;

    // pre-fill the counts with 0
    smemExpertCount[threadIdx.x] = 0;
    __syncthreads();

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    // each thread keeps has some number of "expanded indexes" assigned to it
    // for each of these, we keep the associated expert and offset within expert in registers
    int32_t expertIndexes[MaxExpandedIdxPerThread];
    int32_t expertOffsets[MaxExpandedIdxPerThread];
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
    // time, and branch between a fast path without bound checks and a slow path with bound checks.
    int constexpr IterStride = 4;
    static_assert(MaxExpandedIdxPerThread % IterStride == 0);

    // Define a lambda to avoid code duplication in both branches.
    auto loopBody = [&](int ii, int expandedIdx)
    {
        int32_t expertIdx = params.mPtrExpertIdx[expandedIdx];
        expertIndexes[ii] = expertIdx;
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + expertIdx, 1) : 0;
    };

#pragma unroll
    for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride)
    {
        // Whether it's safe to do multiple iterations without bound checks.
        bool const takeFastPath = (ii0 + IterStride) * numThreadsPerGrid <= expandedIdxSize;
        if (takeFastPath)
        {
#pragma unroll
            for (int32_t jj = 0; jj < IterStride; jj++)
            {
                int const ii = ii0 + jj;
                auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
                loopBody(ii, expandedIdx);
            }
        }
        else
        {
            bool doBreak = false;
#pragma unroll
            for (int32_t jj = 0; jj < IterStride; jj++)
            {
                int const ii = ii0 + jj;
                auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
                if (expandedIdx >= expandedIdxSize)
                {
                    doBreak = true;
                    break;
                }
                loopBody(ii, expandedIdx);
            }
            if (doBreak)
            {
                break;
            }
        }
    }

    // Make histogram (token counts per expert) available to all threads in the block.
    __syncthreads();

    //
    // Each thread now represents one expert
    //

    // Add the local bin count to the common bin count and get a per-CTA offset.
    int32_t const localExpertCount = smemExpertCount[threadIdx.x];
    int32_t const blockExpertOffset = atomicAdd(&params.mPtrExpertCounts[threadIdx.x], localExpertCount);

    // Sync to wait for completion of the histogram reduction.
    grid.sync();

    // Get total count for this expert.
    int32_t count = params.mPtrExpertCounts[threadIdx.x];

    // Note: the scan is redundant in all CTAs, but doing it in only 1 CTA would be worse for latency.

    // Compute the runtime config for projections
    // Whether or not an expert is local is taken into account when smemExpertCount is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    // Strided loop to share this work between blocks.
    int32_t tokensPerTile = params.mAllToAllRouteAct ? params.mNumTokens : count;
    for (int32_t cta = gridBlockIdx; cta < numCta; cta += numBlocks)
    {
        const int32_t localExpertIdx = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
        params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
        params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta] = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
            mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + tokensPerTile);
    }

    // get the padded offset associated with this expert
    const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
    const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);

    // write out padded count
    if (gridBlockIdx == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync())
    {
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    // write expert offsets to shared
    smemExpertOffset[threadIdx.x] = offset + blockExpertOffset;

    // make expert offsets available to all threads
    __syncthreads();

    // trigger the secondary kernel when using PDL
    // We can't do it earlier because FC1 depends on the mPtrCtaIdxXyToBatchIdx,
    // mPtrCtaIdxXyToMnLimit, mPtrNumNonExitingCtas and mPtrTotalNumPaddedTokens
    // TODO: this is not sufficient to ensure visibility in the next kernel!
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }

// each thread has the same "expanded indexes" assigned to it as above
// at this point, we know the final offsets of experts and the offsets within
// experts, which allows writing the final index values
#pragma unroll
    for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ++ii)
    {
        auto expandedIdx = static_cast<int32_t>(gridThreadIdx) + ii * numThreadsPerGrid;
        if (expandedIdx >= expandedIdxSize)
        {
            break;
        }
        auto expertIdx = expertIndexes[ii];
        // check whether this expert is local to our GPU at all
        auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        auto tokenIdx = expandedIdx / params.mTopK;
        auto permutedIdx = isLocalExpert ? int32_t{smemExpertOffset[expertIdx]} + expertOffsets[ii] : int32_t{-1};
        if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
        {
            params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
        }
        if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
        {
            params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
        }
    }
}
#else
__global__ void routingIndicesCoopKernel(KernelParams params)
{
    assert(false && "routingIndicesCoopKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

// Two-step approach (if number of tokens exceed limits of what cluster / cooperative launch
// variants can handle): in order to minimize the amount of data to exchange through global memory,
// we will compute the local histograms in smem twice: the first kernel will get us the total number
// of tokens per expert. The second kernel will use the smem and L2 atomics to get corresponding
// element and tile offsets.
//
// Note: the histogram calculation could also be fused with routingMainKernel, but this might be
// inefficient if we have one CTA per token doing a single global atomic.

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __launch_bounds__(NumThreads) routingIndicesHistogramKernel(KernelParams params)
{
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];

    // For unrolling.
    int32_t constexpr NumEltsPerThread = 8;

    // Pre-fill the counts with 0
    smemExpertCount[threadIdx.x] = 0;
    __syncthreads();

    // Wait on primary grid and trigger secondary kernel.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
        cudaTriggerProgrammaticLaunchCompletion();
    }

    int32_t const expandedIdxSize = params.mNumTokens * params.mTopK;
    int32_t const localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;

    int32_t const gridBlockOffset = blockIdx.x * NumThreads;
    int32_t const gridStride = gridDim.x * NumThreads;

    // Define a lambda to avoid code duplication in branches.
    auto loopBody = [&](int expandedIdx)
    {
        int32_t expertIdx = params.mPtrExpertIdx[expandedIdx];
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        if (isLocalExpert)
        {
            atomicAdd(&smemExpertCount[expertIdx], 1);
        }
    };

    // Grid-stride loop.
    for (int32_t expandedIdx0 = gridBlockOffset * NumEltsPerThread; expandedIdx0 < expandedIdxSize;
         expandedIdx0 += gridStride * NumEltsPerThread)
    {
        // Fast path if bound checks aren't necessary
        if (expandedIdx0 + NumEltsPerThread * NumThreads <= expandedIdxSize)
        {
#pragma unroll
            for (int32_t ii = 0; ii < NumEltsPerThread; ii++)
            {
                int32_t expandedIdx = expandedIdx0 + ii * NumThreads + threadIdx.x;
                loopBody(expandedIdx);
            }
        }
        else
        {
            for (int32_t expandedIdx = expandedIdx0 + threadIdx.x; expandedIdx < expandedIdxSize;
                 expandedIdx += NumThreads)
            {
                loopBody(expandedIdx);
            }
        }
    }
    __syncthreads();

    //
    // Each thread now represents one expert
    //

    // Reduce histograms with atomics.
    int32_t const localExpertCount = smemExpertCount[threadIdx.x];
    atomicAdd(&params.mPtrExpertCounts[threadIdx.x], localExpertCount);
}
#else
__global__ void routingIndicesHistogramKernel(KernelParams params)
{
    assert(false && "routingIndicesHistogramKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __launch_bounds__(NumThreads) routingIndicesOffsetsKernel(KernelParams params)
{
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertTileOffset[NumThreads];
    // needed for the exclusive sum of token offsets
    using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    static constexpr int MaxExpandedIdxPerThread = NumEltsPerOffsetTilePerThread;
    static constexpr int MaxExpandedIdxPerBlock = NumThreads * MaxExpandedIdxPerThread;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

    int32_t const expandedIdxSize = params.mNumTokens * params.mTopK;
    int32_t const numTiles = (expandedIdxSize + MaxExpandedIdxPerBlock - 1) / (MaxExpandedIdxPerBlock);

    // Wait on primary grid.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    // The expert offsets are common to all tiles of all blocks.
    // Load the histogram, scan it and write offsets to shared memory.
    // Note: the scan is redundant in all CTAs. Would it make sense to use an intermediate kernel for
    // the scan, with PDL?

    // Each thread represents one expert. Get total count for this expert.
    int32_t count = params.mPtrExpertCounts[threadIdx.x];

    // Compute the runtime config for projections
    // Whether or not an expert is local is taken into account when the histogram is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    // Get the padded offset associated with this expert
    const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
    const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);

    // Write expert offsets to shared
    smemExpertOffset[threadIdx.x] = offset;
    // Sync to make expert offsets available to all threads.
    __syncthreads();

    // The first block writes out padded count
    if (blockIdx.x == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync())
    {
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    // Strided loop to share this work between blocks.
    int32_t tokensPerTile = params.mAllToAllRouteAct ? params.mNumTokens : count;
    for (int32_t cta = blockIdx.x; cta < numCta; cta += gridDim.x)
    {
        const int32_t localExpertIdx = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
        params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
        params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta] = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
            mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + tokensPerTile);
    }

    //
    // Now loop on indices and compute offsets.
    //

    // Grid-stride loop on 1D "tiles" of input indices.
    for (int32_t tileIdx = blockIdx.x; tileIdx < numTiles; tileIdx += gridDim.x)
    {
        if (tileIdx > 0)
        {
            // Sync for safe reuse of smem buffers.
            __syncthreads();
        }

        // Pre-fill the counts with 0
        smemExpertCount[threadIdx.x] = 0;
        __syncthreads();

        // each thread keeps has some number of "expanded indexes" assigned to it
        // for each of these, we keep the associated expert and offset within expert in registers
        int32_t expertIndexes[MaxExpandedIdxPerThread];
        int32_t expertOffsets[MaxExpandedIdxPerThread];
        auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;

        // Define a lambda to avoid code duplication in branches.
        auto loopBody = [&](int ii, int expandedIdx)
        {
            int32_t expertIdx = params.mPtrExpertIdx[expandedIdx];
            expertIndexes[ii] = expertIdx;
            // check whether this expert is local to our GPU at all and ignore if not
            auto localExpertIdx = expertIdx - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + expertIdx, 1) : 0;
        };

        // For all tiles but the last, all indices are in bounds.
        if (tileIdx < numTiles - 1)
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                loopBody(ii, expandedIdx);
            }
        }
        else
        {
            // For the last tile, we need to exit the loop when out of bounds.
            // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
            // time, and branch between a fast path without bound checks and a slow path with bound checks
            int constexpr IterStride = 4;
            static_assert(MaxExpandedIdxPerThread % IterStride == 0);

#pragma unroll
            for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride)
            {
                // Whether it's safe to do multiple iterations without bound checks.
                bool const takeFastPath
                    = tileIdx * MaxExpandedIdxPerBlock + (ii0 + IterStride) * NumThreads <= expandedIdxSize;
                if (takeFastPath)
                {
#pragma unroll
                    for (int32_t jj = 0; jj < IterStride; jj++)
                    {
                        int const ii = ii0 + jj;
                        auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                        loopBody(ii, expandedIdx);
                    }
                }
                else
                {
                    bool doBreak = false;
#pragma unroll
                    for (int32_t jj = 0; jj < IterStride; jj++)
                    {
                        int const ii = ii0 + jj;
                        auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                        if (expandedIdx >= expandedIdxSize)
                        {
                            doBreak = true;
                            break;
                        }
                        loopBody(ii, expandedIdx);
                    }
                    if (doBreak)
                    {
                        break;
                    }
                }
            }
        }

        // Make local histogram (token counts per expert) available to all threads in the block.
        __syncthreads();

        // Each thread now represents one expert

        // Add the local bin count to the common bin count and get a per-CTA offset. We use the second
        // half of the histogram buffer for this histogram, because the first half already holds the
        // reduced histogram from the previous kernel.
        int32_t const localExpertCount = smemExpertCount[threadIdx.x];
        int32_t const tileExpertOffset
            = atomicAdd(&params.mPtrExpertCounts[NumThreads + threadIdx.x], localExpertCount);

        // Make per-expert tile offsets available to all threads in the block.
        smemExpertTileOffset[threadIdx.x] = tileExpertOffset + smemExpertOffset[threadIdx.x];
        __syncthreads();

        // Add tile offset and element offset and write to global memory.
        auto storeLoopBody = [&](int ii, int expandedIdx)
        {
            int32_t expertIdx = expertIndexes[ii];
            // check whether this expert is local to our GPU at all
            auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            auto tokenIdx = expandedIdx / params.mTopK;
            auto permutedIdx = isLocalExpert ? (expertOffsets[ii] + smemExpertTileOffset[expertIdx]) : int32_t{-1};
            if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
            {
                params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
            }
            if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
            {
                params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
            }
        };
        // Bound checks only in last tile.
        if (tileIdx < numTiles - 1)
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                storeLoopBody(ii, expandedIdx);
            }
        }
        else
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreads + threadIdx.x;
                if (expandedIdx >= expandedIdxSize)
                {
                    break;
                }
                storeLoopBody(ii, expandedIdx);
            }
        }
    }

    // Trigger secondary kernel.
    // Note: this does not guarantee the visibility of prior writes unless the consumer executes a
    // dependency sync.
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
}
#else
__global__ void routingIndicesOffsetsKernel(KernelParams params)
{
    assert(false && "routingIndicesOffsetsKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    TLLM_CHECK_WITH_INFO(data.mPtrExpertIdx != nullptr || data.mPtrPermutedIdxSize != nullptr
            || data.mPtrExpertWeightsFull != nullptr || data.mPtrExpertWeights != nullptr,
        "Routing kernel requires at least one output parameter");
    if (data.mPtrExpandedIdxToPermutedIdx != nullptr || data.mPtrPermutedIdxToTokenIdx != nullptr)
        TLLM_CHECK_WITH_INFO(data.mPtrExpertIdx != nullptr && data.mPtrPermutedIdxSize,
            "If permuted index is required, `mPtrExpertIdx` is also required");
    TLLM_CHECK_WITH_INFO(!data.mUseRoutingSoftmax, "Routing with softmax not implemented yet");
    TLLM_CHECK_WITH_INFO(data.mNumLimitedGroups <= MaxNumTopGroups, "Routing kernel expects <= %d top groups, got %d",
        MaxNumTopGroups, data.mNumLimitedGroups);
    TLLM_CHECK_WITH_INFO(data.mTopK <= MaxNumTopExperts, "Routing kernel expects topK experts <= %d, got %d",
        MaxNumTopExperts, data.mTopK);
    TLLM_CHECK_WITH_INFO(data.mTopK <= WarpSize, "Routing kernel expects top K <= warp size, got %d", data.mTopK);
    TLLM_CHECK_WITH_INFO(data.mTopK * data.mNumLimitedGroups <= WarpSize,
        "Routing kernel expects top K * top groups <= warp size (for now), got %d * %d", data.mTopK,
        data.mNumLimitedGroups);
    TLLM_CHECK_WITH_INFO(data.mNumExperts >= MaxNumTopExperts, "Routing kernel expects %d to be at most #experts %d",
        MaxNumTopExperts, data.mNumExperts);
    TLLM_CHECK_WITH_INFO(data.mNumExperts <= NumThreads, "Routing kernel expects #experts %d  <= #threads %d",
        data.mNumExperts, NumThreads);
    TLLM_CHECK_WITH_INFO(data.mNumExpertGroups >= data.mNumLimitedGroups,
        "Routing kernel expects top groups %d to be limited by #expert groups %d", data.mNumLimitedGroups,
        data.mNumExpertGroups);
    if (data.mNumExpertGroups > 1)
    {
        TLLM_CHECK_WITH_INFO(data.mNumExpertGroups <= NumWarps,
            "Routing kernel expects #experts groups %d to be <= #warps %d", data.mNumExpertGroups, NumWarps);
        TLLM_CHECK_WITH_INFO(data.mNumExperts % data.mNumExpertGroups == 0,
            "Routing kernel expects #experts %d to be a multiple of #expert groups %d", data.mNumExperts,
            data.mNumExpertGroups);
        TLLM_CHECK_WITH_INFO(data.mNumExperts / data.mNumExpertGroups <= WarpSize,
            "Routing kernel expects #experts per group <= warp size, got %d", data.mNumExperts / data.mNumExpertGroups);
    }
    else
    {
        TLLM_CHECK_WITH_INFO(data.mNumExperts <= WarpSize * MaxNumTopGroups,
            "Routing kernel expects #experts %d <= WarpSize * MaxNumTopGroups %d", data.mNumExperts,
            WarpSize * MaxNumTopGroups);
        TLLM_CHECK_WITH_INFO(
            data.mTopK <= NumWarps, "Routing kernel expects top K %d to be <= #warps %d", data.mTopK, NumWarps);
    }
    TLLM_CHECK_WITH_INFO(
        data.mNumExperts % 4 == 0, "Routing kernel expects #experts %d to be a multiple of 4.", data.mNumExperts);
    TLLM_CHECK_WITH_INFO(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got %d", data.mPaddingLog2);
    int const numBlocks = data.mNumTokens;

    if (data.mPtrExpertWeightsFull != nullptr)
    {
        auto localExpertExtent = data.mNumLocalExperts << data.mLocalExpertsStrideLog2;
        // note: we set a value of 0 here, s.t. even if the routing happens,
        // it will be ignored / not given any weight
        TLLM_CUDA_CHECK(hipMemsetAsync(
            data.mPtrExpertWeightsFull, 0, localExpertExtent * data.mNumTokens * sizeof(float), (hipStream_t) stream));
    }

    /*  disable memset(-1) for permuted_idx_to_token_idx for performance
        if (data.mPtrPermutedIdxToTokenIdx != nullptr)
        {
            // need to set all values to -1 before running the kernel
            auto maxPermutedSize
                = data.mNumTokens * data.mTopK + (data.mNumExperts << data.mPaddingLog2) - data.mNumExperts;
            // note that a value of -1 per byte works for any size of signed integer
            // to set each full value to the logical value -1
            TLLM_CUDA_CHECK(hipMemsetAsync(data.mPtrPermutedIdxToTokenIdx, -1,
                static_cast<size_t>(maxPermutedSize) * sizeof(int32_t), (hipStream_t) stream));
        }
    */

    bool const useSingleCluster = data.mNumTokens <= 1024;
    if (!useSingleCluster)
    {
        // Reset the global histograms (not used in single-cluster code path).
        // Cover both for the cooperative and two-kernel code paths.
        TLLM_CUDA_CHECK(hipMemsetAsync(
            data.mPtrExpertCounts, 0, static_cast<size_t>(2 * NumThreads) * sizeof(int32_t), (hipStream_t) stream));
    }

    // Number of blocks we can use in the cooperative kernel
    // The number of blocks must be:
    //   >= ⌈(numTokens * topK) / (MaxExpandedIdxPerThread * NumThreads)⌉
    //   <= numSms, assuming an occupancy of 1 block/SM
    //
    // If too small for the given numTokens, fall back to the less performant two-step method.
    //
    // The upper bound is a strict requirement. The number of blocks should be determined by querying
    // the device properties, or conservatively low.
    // /!\ The following number is not portable!! (but works on H100 and B200)
    int const numBlocksCoop = 128;

    // Maximum number of tokens supported by the kernel using a cooperative launch.
    int const maxTokensCoop = (numBlocksCoop * NumThreads * 64) / data.mTopK;
    LAUNCH_EXPW_ONLY_GROUPS(data,
        /*coopLaunch=*/false, routingMainKernel, numBlocks, NumThreads,
        /*smemSize=*/0, // No dynamic smem
        stream);

    if (data.mPtrPermutedIdxSize != nullptr)
    {
        if (useSingleCluster)
        {
            LAUNCH_EXPW_ONLY_GROUPS(data,
                /*coopLaunch=*/false, routingIndicesClusterKernel, NumBlocksPerCluster, NumThreads,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
        else if (data.mNumTokens <= maxTokensCoop)
        {
            LAUNCH_EXPW_ONLY_GROUPS(data,
                /*coopLaunch=*/true, routingIndicesCoopKernel, numBlocksCoop, NumThreads,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
        else
        {
            const int32_t expandedIdxSize = data.mNumTokens * data.mTopK;

            const int32_t histogramEltsPerBlock = 8 * NumThreads;
            const int32_t offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreads;

            // Limit grid size (both kernels use a grid-stride loop).
            const int32_t maxNumBlocks = 1024;

            int const numBlocksHistogram
                = std::min((expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
            int const numBlocksOffsets
                = std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

            LAUNCH_EXPW_ONLY_GROUPS(data,
                /*coopLaunch=*/false, routingIndicesHistogramKernel, numBlocksHistogram, NumThreads,
                /*smemSize=*/0, // No dynamic smem
                stream);
            LAUNCH_EXPW_ONLY_GROUPS(data,
                /*coopLaunch=*/false, routingIndicesOffsetsKernel, numBlocksOffsets, NumThreads,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace routing

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace routingLlama4
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace tg = batchedGemm::trtllm::gen;
namespace cg = cooperative_groups;

////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 1024;
static constexpr int NumThreadsHist = 256;
static constexpr int NumBlocksPerCluster = 8;
static constexpr int WarpSize = 32;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int NumWarpsHist = NumThreadsHist / WarpSize;
static constexpr int NumTopExperts = 1;
static constexpr int MaxNumExperts = 128;
static constexpr int MaxNumTokensSingleCluster = NumBlocksPerCluster * NumThreads;
static constexpr int MaxNumTokensSingleClusterScores = NumBlocksPerCluster * NumWarps;
static constexpr int WarpKernelSmemStride = 33;
// with further optimization to `routingIndicesWarpKernel`, this limit may
// increase. For now, it is a good cut-off point for when the block-wise
// operations are more efficient end-to-end.
static constexpr int WarpKernelMaxNumTokens = 4;

// Performance tuning knob.
static constexpr int NumEltsPerOffsetTilePerThread = 8;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 1000 && defined(__CUDA_ARCH_FEAT_SM100_ALL))
#define TLLM_GEN_ENABLE_FAST_REDUX
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename TypeExpW_>
struct TopKRedType
{
    using TypeExpW = TypeExpW_;
    static_assert(std::is_same_v<TypeExpW, float> || std::is_same_v<TypeExpW, cutlass::bfloat16_t>,
        "Top K reduction only implemented for float and Bf16");
    using TypeCmp = std::conditional_t<sizeof(TypeExpW) >= 4, double, float>;
    static constexpr int64_t Mask64 = 0x000000000000FFFF;
    static constexpr int32_t Mask32 = 0x0000FFFF;

    TypeCmp compVal;

    static __host__ __device__ inline TypeCmp makeCmpVal(TypeExpW val, int32_t idx = 0)
    {
        auto cmpVal = TypeCmp{val};
        TypeCmp cmpValWithIdx;
        if constexpr (sizeof(TypeExpW) >= 4)
        {
            auto cmpValIdx64 = reinterpret_cast<int64_t&>(cmpVal) | (Mask64& int64_t{idx});
            cmpValWithIdx = reinterpret_cast<TypeCmp&>(cmpValIdx64);
        }
        else
        {
            auto cmpValIdx32 = reinterpret_cast<int32_t&>(cmpVal) | (Mask32 & idx);
            cmpValWithIdx = reinterpret_cast<TypeCmp&>(cmpValIdx32);
        }
        return cmpValWithIdx;
    }

    static __host__ __device__ inline void unpack(TypeExpW& val, int32_t& idx, TypeCmp cmp)
    {
        if constexpr (sizeof(TypeExpW) >= 4)
        {
            idx = static_cast<int32_t>(reinterpret_cast<int64_t&>(cmp) & Mask64);
            auto val64 = reinterpret_cast<int64_t&>(cmp) & ~Mask64;
            val = static_cast<float>(reinterpret_cast<double&>(val64));
        }
        else
        {
            idx = reinterpret_cast<int32_t&>(cmp) & Mask32;
            auto val32 = reinterpret_cast<int32_t&>(cmp) >> 16;
            val = TypeExpW::bitcast(reinterpret_cast<uint16_t&>(val32));
        }
    }

    __host__ __device__ TopKRedType() = default;

    __host__ __device__ TopKRedType(TypeExpW val, int32_t idx)
        : compVal(makeCmpVal(val, idx))
    {
    }

    __host__ __device__ operator TypeCmp() const noexcept
    {
        return compVal;
    }

    __device__ inline TypeCmp reduce(cg::thread_block_tile<WarpSize> const& warp)
    {
#if defined(TLLM_GEN_ENABLE_FAST_REDUX)
        static constexpr bool UseCg = false;
#else
        static constexpr bool UseCg = true;
#endif
        if constexpr (UseCg || sizeof(TypeExpW) >= 4)
        {
            return cg::reduce(warp, compVal, cg::greater<TypeCmp>{});
        }
        else
        {
            float result;
            asm("redux.sync.max.f32 %0, %1, 0xffffffff;\n" : "=f"(result) : "f"(compVal));
            return result;
        }
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

static __device__ inline float sigmoid_accurate(float x)
{
    return 0.5f * tanhf(0.5f * x) + 0.5f;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K_, bool Enable_>
struct TopKIdx
{
    // by default, empty
};

template <int K_>
struct TopKIdx<K_, true>
{
    static constexpr int K = K_;
    int32_t val[K];
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K, typename Type>
__device__ void reduceTopK(cg::thread_block_tile<WarpSize> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type value, int32_t idx, Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WarpSize, "Top K must have K < WarpSize");
    using RedType = TopKRedType<Type>;
    RedType topK{value, idx};
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        topK = kk > 0 && packedMax == topK.compVal ? RedType{minValue, idx} : topK;
        // get the next largest value
        packedMax = topK.reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

#define TOPK_SWAP(I, J)                                                                                                \
    {                                                                                                                  \
        auto pairMin = min(topK[I].compVal, topK[J].compVal);                                                          \
        auto pairMax = max(topK[I].compVal, topK[J].compVal);                                                          \
        topK[I].compVal = pairMax;                                                                                     \
        topK[J].compVal = pairMin;                                                                                     \
    }

template <int K, typename Type, int N, bool IsSorted = false>
__device__ void reduceTopK(cg::thread_block_tile<WarpSize> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type (&value)[N], int32_t (&idx)[N], Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WarpSize, "Top K must have K < WarpSize");
    static_assert(N > 0, "Top K must have N > 1");
    static_assert(N <= K, "Top K must have N < K");
    using RedType = TopKRedType<Type>;
    RedType topK[N];
#pragma unroll
    for (int nn = 0; nn < N; ++nn)
        topK[nn] = RedType{value[nn], idx[nn]};
    if constexpr (!IsSorted)
    {
        TOPK_SWAP(0, 2);
        TOPK_SWAP(1, 3);

        TOPK_SWAP(0, 1);
        TOPK_SWAP(2, 3);

        TOPK_SWAP(1, 2);
    }
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        bool update = kk > 0 && packedMax == topK[0].compVal;
#pragma unroll
        for (int nn = 0; nn < N; ++nn)
        {
            topK[nn] = update && nn == N - 1 ? RedType{minValue, idx[nn]} : update ? topK[nn + 1] : topK[nn];
        }
        // get the next largest value
        packedMax = topK[0].reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

#undef TOPK_SWAP

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T mulLog2(T a, T bLog2)
{
    return a << bLog2;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T divUpLog2(T a, T bLog2)
{
    return ((a + (1 << bLog2) - 1) >> bLog2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T divUpMulLog2(T a, T bLog2)
{
    return mulLog2<T>(divUpLog2<T>(a, bLog2), bLog2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ __device__ constexpr int32_t getBits(int32_t value, int idx)
{
    int mask = idx == 0 ? 0x000000FF : idx == 1 ? 0x0000FF00 : idx == 2 ? 0x00FF0000 : 0xFF000000;
    return (value & mask) >> (idx * 8);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <bool IsZero = false>
__host__ __device__ constexpr void setBits(int32_t& value, int32_t newBits, int idx)
{
    if constexpr (!IsZero)
    {
        int mask = idx == 0 ? 0xFFFFFF00 : idx == 1 ? 0xFFFF00FF : idx == 2 ? 0xFF00FFFF : 0x00FFFFFF;
        value &= mask;
    }
    value |= (newBits << (idx * 8));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __launch_bounds__(WarpSize) routingIndicesWarpKernel(KernelParams params)
{
    // types used in this kernel
    using TypeExpW = typename KernelParams::TypeExpW;
    using TypePacked = PackedScoreIdx<TypeExpW>;
    // use the default cub warp-scan, with shfl
    using Scan = hipcub::WarpScan<int32_t>;
    __shared__ typename Scan::TempStorage tempStorage;

    // each thread encodes 4 experts in one `int32_t`. The assumption is that
    // we don't have more than 127 tokens, but `WarpKernelMaxNumTokens` must be
    // smaller than that because other approaches will be more efficient for
    // 127 tokens.
    static constexpr int ExpertsPerThread = sizeof(int32_t);
    static_assert(WarpKernelMaxNumTokens <= 127);
    // this is a full table of which token is routed to which expert.
    // the assumption here is that there are no more than 128 experts.
    // we use a stride of 33 instead of 32 to avoid shared memory bank conflicts.
    __shared__ int32_t __attribute((aligned(128)))
    smemExpertTokenCountFull[WarpKernelMaxNumTokens][WarpKernelSmemStride];
    static_assert(WarpKernelSmemStride == WarpSize + 1);
    static_assert(MaxNumExperts / sizeof(int32_t) <= WarpSize);

    // values needed for the top-1 reduction, if required
    TypeExpW minScore = TypeExpW{-INFINITY};
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

#pragma unroll
    for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens; ++tokenIdx)
    {
        // reset full shared memory field to 0
        smemExpertTokenCountFull[tokenIdx][threadIdx.x] = 0;
    }
    __syncwarp();

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    if (params.mPtrScores != nullptr)
    {
        // if we use `mPtrScores` as input, we need to perform the top-1 reduction
        // for each token, we load the scores then use `reduceTopK` for this.
        // each thread works on 4 experts, so a local reduction is done before
        for (int tokenIdx = 0; tokenIdx < params.mNumTokens; ++tokenIdx)
        {
            auto scoreOffset = tokenIdx * params.mNumExperts;
            // local reduction to get the best score for our 4 experts
            TypeExpW maxScore = minScore;
            int32_t maxExpertIdx{-1};
#pragma unroll
            for (int ii = 0; ii < ExpertsPerThread; ++ii)
            {
                auto expertIdx = ii * WarpSize + threadIdx.x;
                auto newScore = expertIdx < params.mNumExperts ? params.mPtrScores[scoreOffset + expertIdx] : minScore;
                // note: use `>=` s.t. highest index always wins, just like in `reduceTopK`
                maxExpertIdx = newScore >= maxScore ? expertIdx : maxExpertIdx;
                maxScore = newScore >= maxScore ? newScore : maxScore;
            }
            int32_t warpMaxExpertIdx[NumTopExperts];
            TypeExpW warpMaxScore[NumTopExperts];
            // warp-wide reduction to get the best score for all experts
            reduceTopK(warp, warpMaxScore, warpMaxExpertIdx, maxScore, maxExpertIdx, minScore);
            if (cute::elect_one_sync())
            {
                // one thread updates the count linking token to chosen expert
                auto expertTokenCount = 0;
                setBits</* IsZero= */ true>(expertTokenCount, 1, warpMaxExpertIdx[0] % ExpertsPerThread);
                smemExpertTokenCountFull[tokenIdx][warpMaxExpertIdx[0] / ExpertsPerThread] = expertTokenCount;
                // we also compute the final score here and write it out if required
                auto finalScore = TypeExpW{sigmoid_accurate(float{warpMaxScore[0]})};
                if (params.mPtrExpertWeights != nullptr)
                {
                    params.mPtrExpertWeights[tokenIdx] = finalScore;
                }
            }
        }
    }
    else
    {
        // if we do not have `mPtrScores` as input, we expect that `mPtrExpertWeights`
        // contains the top-1 packed score and index already.
        // Each thread represents a token here, and we extract the relevant score
        // The assumption is that the #tokens is limited by warp-size
        static_assert(WarpKernelMaxNumTokens <= WarpSize);
        TypePacked scoreIdx = threadIdx.x < params.mNumTokens ? params.mPtrExpertIdx[threadIdx.x] : TypePacked{};
        int32_t expertTokenCount = 0;
        setBits</* IsZero= */ true>(expertTokenCount, 1, scoreIdx.idx % ExpertsPerThread);
        if (threadIdx.x < params.mNumTokens)
        {
            smemExpertTokenCountFull[threadIdx.x][scoreIdx.idx / ExpertsPerThread] = expertTokenCount;
        }
        // we also compute the final score here and write it out if required
        auto finalScore = TypeExpW{sigmoid_accurate(float{scoreIdx.score})};
        if (params.mPtrExpertWeights != nullptr && threadIdx.x < params.mNumTokens)
        {
            params.mPtrExpertWeights[threadIdx.x] = finalScore;
        }
    }

    // make the full table available to all threads
    __syncwarp();

    // at this point, each thread keeps a count of its 4 assigned experts in
    // `expertCount`, as well as the offsets for all tokens w.r.t. these 4 experts
    // in `expertOffset`.
    int32_t expertCount = 0;
    int32_t expertOffset[WarpKernelMaxNumTokens + 1];
#pragma unroll
    for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens + 1; ++tokenIdx)
    {
        if (tokenIdx > params.mNumTokens)
            break;
        // simple reduction for `expertCount`, and scan for `expertOffset`
        auto expertTokenCount = tokenIdx < params.mNumTokens ? smemExpertTokenCountFull[tokenIdx][threadIdx.x] : 0;
        expertOffset[tokenIdx] = expertCount;
        expertCount += expertTokenCount;
    }

    // at this point, we are ready for the scan across all experts to get the
    // thread-wise offsets across experts
    // first, we need to reduce across our 4 experts into `numCta`
    int32_t numCta = 0;
#pragma unroll
    for (int ii = 0; ii < ExpertsPerThread; ++ii)
    {
        auto count = getBits(expertCount, ii);
        numCta += divUpLog2<int32_t>(count, params.mPaddingLog2);
    }
    // second, we perform the exclusive sum across the warp
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    // finally, we perform a scan across our local experts, starting with the
    // warp-wide scan result (`ctaOffset`)
    auto ctaOffsetExp = ctaOffset;
#pragma unroll
    for (int ii = 0; ii < ExpertsPerThread; ++ii)
    {
        auto count = getBits(expertCount, ii);
        auto finalNumCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
        auto expertIdx = threadIdx.x * ExpertsPerThread + ii;
        // during the scan for expert offsets, we can already write out
        // both `mPtrCtaIdxXyToBatchIdx` and `mPtrCtaIdxXyToMnLimit`
        for (int cta = 0; cta < finalNumCta; ++cta)
        {
            params.mPtrCtaIdxXyToBatchIdx[ctaOffsetExp + cta] = expertIdx;
            params.mPtrCtaIdxXyToMnLimit[ctaOffsetExp + cta]
                = min(mulLog2<int32_t>(ctaOffsetExp + cta + 1, params.mPaddingLog2),
                    mulLog2<int32_t>(ctaOffsetExp, params.mPaddingLog2) + count);
        }
        ctaOffsetExp += finalNumCta;
    }

    // at this point, we can write out padded count from the warp-aggregate
    if (cute::elect_one_sync())
    {
        const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

#if !defined(PDL_PROFILE) || PDL_PROFILE == 0
    // we can trigger the next kernel at this point
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
#endif

    // at this point, all values for offsets are ready, except the final offsets
    // within the padded index (`permutedIdx`)
    // for this, we perform a scan similar to the one directly after the warp-scan:
    // here, we keep the local offset for each of the thread's experts in a field
    // of registers
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    int32_t finalExpertOffset[ExpertsPerThread];
    finalExpertOffset[0] = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);
#pragma unroll
    for (int ii = 1; ii < ExpertsPerThread; ++ii)
    {
        finalExpertOffset[ii]
            = finalExpertOffset[ii - 1] + divUpMulLog2<int32_t>(getBits(expertCount, ii - 1), params.mPaddingLog2);
    }

#pragma unroll
    for (int tokenIdx = 0; tokenIdx < WarpKernelMaxNumTokens; ++tokenIdx)
    {
        // at this point, we can calculate the final index:
        // we simply loop over all tokens, and all experts assigned to this thread.
        // For each pair, we determine whether that token was routed to that expert
        // based on whether the offset for that token changed.
        // we can then easily compute the final `expertIdx` and `permutedIdx` relative
        // to this token and expert, and write them out.
        if (tokenIdx >= params.mNumTokens)
            break;

#pragma unroll
        for (int ii = 0; ii < ExpertsPerThread; ++ii)
        {
            // determine whether the offset for this expert and token changes
            auto localOffsetToken = getBits(expertOffset[tokenIdx], ii);
            auto isTokenRouted = getBits(expertOffset[tokenIdx + 1], ii) > localOffsetToken;
            // the expert index of this expert
            auto expertIdx = threadIdx.x * ExpertsPerThread + ii;
            auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            // the permuted index: we add the local offset relative to this expert and token
            // to the global offset from the scan for this expert
            auto permutedIdx = isLocalExpert ? finalExpertOffset[ii] + localOffsetToken : int32_t{-1};
            // write out `mPtrExpandedIdxToPermutedIdx` if required
            if (params.mPtrExpandedIdxToPermutedIdx != nullptr && isTokenRouted)
            {
                params.mPtrExpandedIdxToPermutedIdx[tokenIdx] = permutedIdx;
            }
            // write out `mPtrPermutedIdxToTokenIdx` if required
            if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert && isTokenRouted)
            {
                params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
            }
        }
    }
}
#else
__global__ void routingIndicesWarpKernel(KernelParams params)
{
    assert(false && "routingIndicesWarpKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params)
{
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
    // number of tokens/expanded idx is bounded by total number of warps
    using TypeExpW = typename KernelParams::TypeExpW;
    using TypePacked = PackedScoreIdx<TypeExpW>;
    __shared__ TypePacked __attribute((aligned(128))) smemPackedScoreIdx[NumWarps];
    // Needed for the exclusive sum of token offsets.
    // Note: the scan might include more bins than needed, with bin counts of 0 to pad
    using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    // Number of threads in the cluster.
    static constexpr int NumThreadsPerCluster = NumThreads * NumBlocksPerCluster;
    // same as max num tokens
    static constexpr int MaxExpandedIdxPerThread
        = (MaxNumTokensSingleCluster * NumTopExperts + NumThreadsPerCluster - 1) / NumThreadsPerCluster;

    uint32_t const clusterBlockRank = blockIdx.x;
    uint32_t const clusterThreadIdx = NumThreads * clusterBlockRank + threadIdx.x;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
    int32_t const laneIdx = cutlass::arch::LaneId();

    auto expandedIdxSize = params.mNumTokens * NumTopExperts;
    // TODO(mjoux): expand to more tokens (possibly)
    auto warpTokenIdx = clusterBlockRank * NumWarps + warpIdx;
    auto scoreOffset = warpTokenIdx * params.mNumExperts;
    bool validToken = warpTokenIdx < params.mNumTokens;
    TypeExpW minScore = TypeExpW{-INFINITY};

    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

    // pre-fill the counts with 0
    if (threadIdx.x < params.mNumExperts)
    {
        smemExpertCount[threadIdx.x] = 0;
    }
    __syncthreads();

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    if (params.mPtrScores != nullptr)
    {
        TypeExpW maxScore = minScore;
        int32_t maxExpertIdx{-1};
        // in this case, each warp represents a token
        // we then exchange all token max scores, s.t. afterwards, each thread
        // represents a token
        if (validToken)
        {
#pragma unroll
            for (int i = 0; i < MaxNumExperts / WarpSize; ++i)
            {
                auto expertIdx = i * WarpSize + laneIdx;
                auto newScore = expertIdx < params.mNumExperts ? params.mPtrScores[scoreOffset + expertIdx] : minScore;
                // note: use `>=` s.t. highest index always wins, just like in `reduceTopK`
                maxExpertIdx = newScore >= maxScore ? expertIdx : maxExpertIdx;
                maxScore = newScore >= maxScore ? newScore : maxScore;
            }
            int32_t warpMaxExpertIdx[NumTopExperts];
            TypeExpW warpMaxScore[NumTopExperts];
            reduceTopK(warp, warpMaxScore, warpMaxExpertIdx, maxScore, maxExpertIdx, minScore);
            if (cute::elect_one_sync())
            {
                TypePacked packedScore{warpMaxScore[0], static_cast<int16_t>(warpMaxExpertIdx[0])};
                smemPackedScoreIdx[warpIdx] = packedScore;
            }
        }
        // make packed scores available to all threads in cluster
        __cluster_barrier_arrive();
        __cluster_barrier_wait();
    }

    // each thread keeps some number of "expanded indexes" assigned to it
    // note that expanded indexes simply represent tokens here.
    // for each of these, we keep the associated expert and offset within expert in registers
    int32_t expertIndexes[MaxExpandedIdxPerThread];
    int32_t expertOffsets[MaxExpandedIdxPerThread];
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
    // time, and branch between a fast path without bound checks and a slow path with bound checks.
    // TODO(mjoux): potentially add this back for perf tuning
    // int constexpr IterStride = 4;
    // static_assert(MaxExpandedIdxPerThread % IterStride == 0);

    // Define a lambda to avoid code duplication in both branches.
    auto loopBody = [&](int ii, int expandedIdx)
    {
        TypePacked scoreIdx;
        if (params.mPtrScores != nullptr)
        {
            TypePacked const* remoteSmem
                = cg::cluster_group::map_shared_rank(smemPackedScoreIdx, expandedIdx / NumWarps);
            scoreIdx = remoteSmem[expandedIdx % NumWarps];
        }
        else
        {
            scoreIdx = params.mPtrExpertIdx[expandedIdx];
        }
        expertIndexes[ii] = scoreIdx.idx;
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = scoreIdx.idx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + scoreIdx.idx, 1) : 0;
        auto finalScore = TypeExpW{sigmoid_accurate(float{scoreIdx.score})};
        if (params.mPtrExpertWeights != nullptr)
        {
            params.mPtrExpertWeights[expandedIdx] = finalScore;
        }
    };

    if (clusterThreadIdx < expandedIdxSize)
    {
        loopBody(0, clusterThreadIdx);
    }

    // Make local histogram (token counts per expert) available to all threads in the cluster.
    __cluster_barrier_arrive();
    __cluster_barrier_wait();

    //
    // Each thread now represents one expert
    //

    // Total number of tokens for this expert.
    int32_t count = 0;
    // Per-expert offset for this block.
    int32_t blockExpertOffset = 0;

    if (threadIdx.x < params.mNumExperts)
    {
        // Get the histogram bin from each rank for this expert.
        int32_t expertCounts[NumBlocksPerCluster];
#pragma unroll
        for (int rank = 0; rank < NumBlocksPerCluster; rank++)
        {
            int32_t const* remoteSmem = cg::cluster_group::map_shared_rank(smemExpertCount, rank);
            expertCounts[rank] = rank * NumWarps < params.mNumTokens ? remoteSmem[threadIdx.x] : 0;
        }

        // Compute an exclusive prefix sum of the block-local count.
#pragma unroll
        for (int rank = 0; rank < NumBlocksPerCluster; rank++)
        {
            if (rank == clusterBlockRank)
            {
                blockExpertOffset = count;
            }
            count += expertCounts[rank];
        }
    }

    // Arrive: we do not access distributed shared memory after this point.
    __cluster_barrier_arrive();

    // Compute the runtime config for projections
    // Weather or not an expert is local is taken into account when smemExpertCount is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    if (threadIdx.x < params.mNumExperts)
    {
        // Strided loop to share this work between blocks.
        for (int32_t cta = clusterBlockRank; cta < numCta; cta += NumBlocksPerCluster)
        {
            const int32_t localExpertIdx
                = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
            params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
            params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta]
                = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
                    mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + count);
        }

        // get the padded offset associated with this expert
        const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);

        // write expert offsets to shared
        smemExpertOffset[threadIdx.x] = offset + blockExpertOffset;
    }

    // write out padded count
    if (clusterBlockRank == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync())
    {
        const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    // make expert offsets available to all threads
    __syncthreads();

    // Wait: we cannot exit while other blocks may be accessing the current block's shared memory.
    // Note: I observed a perf benefit to doing this before the final loop so the compiler can
    // implement break with EXIT.
    __cluster_barrier_wait();

    // trigger the secondary kernel when using PDL
    // We can't do it earlier because FC1 depends on the mPtrCtaIdxXyToBatchIdx,
    // mPtrCtaIdxXyToMnLimit, mPtrNumNonExitingCtas and mPtrTotalNumPaddedTokens
    // TODO: this is not sufficient to ensure visibility in the next kernel!
#if !defined(PDL_PROFILE) || PDL_PROFILE == 0
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
#endif

    // each thread has the same "expanded indexes" assigned to it as above
    // at this point, we know the final offsets of experts and the offsets within
    // experts, which allows writing the final index values

#pragma unroll
    for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ++ii)
    {
        auto expandedIdx = static_cast<int32_t>(clusterThreadIdx) + ii * NumThreadsPerCluster;
        if (expandedIdx >= expandedIdxSize)
        {
            break;
        }
        auto expertIdx = expertIndexes[ii];
        // check whether this expert is local to our GPU at all
        auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        auto tokenIdx = expandedIdx / NumTopExperts;
        auto permutedIdx = isLocalExpert ? int32_t{smemExpertOffset[expertIdx]} + expertOffsets[ii] : int32_t{-1};
        if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
        {
            params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
        }
        if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
        {
            params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
        }
    }
}
#else
__global__ void routingIndicesClusterKernel(KernelParams params)
{
    assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

// this kernel is needed in case we have scores as input for the histogram kernel
template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __launch_bounds__(NumThreadsHist) routingIndicesHistogramScoresKernel(KernelParams params)
{
    using TypeExpW = typename KernelParams::TypeExpW;
    using TypeExpWVec = std::conditional_t<sizeof(TypeExpW) == 2, float2, float4>;
    using TypePacked = PackedScoreIdx<TypeExpW>;
    static constexpr int VecSize = MaxNumExperts / WarpSize;
    // we assume that #experts is a multiple of 4, so VecSize must be 4.
    static_assert(VecSize == 4);

    int32_t const laneIdx = cutlass::arch::LaneId();
    int32_t const warpIdx = threadIdx.x / WarpSize;
    int32_t const globalWarpIdx = blockIdx.x * NumWarpsHist + warpIdx;
    int32_t const globalWarpStride = gridDim.x * NumWarpsHist;
    TypeExpW minScore = TypeExpW{-INFINITY};
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

    // Wait on primary grid and trigger secondary kernel.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
        cudaTriggerProgrammaticLaunchCompletion();
    }

    // in this case, each warp represents a token, and we use a grid-stride loop
    // over all warps/tokens
    for (int tokenIdx = globalWarpIdx; tokenIdx < params.mNumTokens; tokenIdx += globalWarpStride)
    {
        TypeExpW maxScore = minScore;
        int32_t maxExpertIdx{-1};
        auto scoreOffset = (tokenIdx * params.mNumExperts) / VecSize + laneIdx;

        TypeExpW allScores[VecSize];
        auto* ptrAllScores = reinterpret_cast<TypeExpWVec const*>(params.mPtrScores);
        *reinterpret_cast<TypeExpWVec*>(allScores) = ptrAllScores[scoreOffset];

#pragma unroll
        for (int i = 0; i < VecSize; ++i)
        {
            auto expertIdx = laneIdx * VecSize + i;
            auto newScore = expertIdx < params.mNumExperts ? allScores[i] : minScore;
            // note: use `>=` s.t. highest index always wins, just like in `reduceTopK`
            maxExpertIdx = newScore >= maxScore ? expertIdx : maxExpertIdx;
            maxScore = newScore >= maxScore ? newScore : maxScore;
        }
        int32_t warpMaxExpertIdx[NumTopExperts];
        TypeExpW warpMaxScore[NumTopExperts];
        reduceTopK(warp, warpMaxScore, warpMaxExpertIdx, maxScore, maxExpertIdx, minScore);
        if (cute::elect_one_sync())
        {
            TypePacked packedScore{warpMaxScore[0], static_cast<int16_t>(warpMaxExpertIdx[0])};
            params.mPtrExpertIdx[tokenIdx] = packedScore;
        }
    }
}
#else
__global__ void routingIndicesHistogramScoresKernel(KernelParams params)
{
    assert(false && "routingIndicesHistogramScoresKernel is only supported on SM90+ architectures");
}
#endif

// Two-step approach (if number of tokens exceed limits of what cluster / cooperative launch
// variants can handle): in order to minimize the amount of data to exchange through global memory,
// we will compute the local histograms in smem twice: the first kernel will get us the total number
// of tokens per expert. The second kernel will use the smem and L2 atomics to get corresponding
// element and tile offsets.
//
// Note: the histogram calculation could also be fused with routingMainKernel, but this might be
// inefficient if we have one CTA per token doing a single global atomic.
template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __launch_bounds__(NumThreadsHist) routingIndicesHistogramKernel(KernelParams params)
{
    using TypeExpW = typename KernelParams::TypeExpW;
    using TypePacked = PackedScoreIdx<TypeExpW>;
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreadsHist];

    // For unrolling.
    uint32_t constexpr NumEltsPerThread = 8;

    // Pre-fill the counts with 0
    if (threadIdx.x < params.mNumExperts)
    {
        smemExpertCount[threadIdx.x] = 0;
    }
    __syncthreads();

    // Wait on primary grid and trigger secondary kernel.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
        cudaTriggerProgrammaticLaunchCompletion();
    }

    uint32_t const expandedIdxSize = params.mNumTokens * NumTopExperts;
    uint32_t const localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;

    uint32_t const gridBlockOffset = blockIdx.x * NumThreadsHist;
    uint32_t const gridStride = gridDim.x * NumThreadsHist;

    // Define a lambda to avoid code duplication in branches.
    auto loopBody = [&](int expandedIdx)
    {
        TypePacked scoreIdx = params.mPtrExpertIdx[expandedIdx];
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = scoreIdx.idx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        if (isLocalExpert)
        {
            atomicAdd(&smemExpertCount[scoreIdx.idx], 1);
        }
        auto finalScore = TypeExpW{sigmoid_accurate(float{scoreIdx.score})};
        if (params.mPtrExpertWeights != nullptr)
        {
            params.mPtrExpertWeights[expandedIdx] = finalScore;
        }
    };

    // Grid-stride loop.
    for (uint32_t expandedIdx0 = gridBlockOffset * NumEltsPerThread; expandedIdx0 < expandedIdxSize;
         expandedIdx0 += gridStride * NumEltsPerThread)
    {
        // Fast path if bound checks aren't necessary
        if (expandedIdx0 + NumEltsPerThread * NumThreadsHist <= expandedIdxSize)
        {
#pragma unroll
            for (uint32_t ii = 0; ii < NumEltsPerThread; ii++)
            {
                uint32_t expandedIdx = expandedIdx0 + ii * NumThreadsHist + threadIdx.x;
                loopBody(expandedIdx);
            }
        }
        else
        {
            for (uint32_t expandedIdx = expandedIdx0 + threadIdx.x; expandedIdx < expandedIdxSize;
                 expandedIdx += NumThreadsHist)
            {
                loopBody(expandedIdx);
            }
        }
    }
    __syncthreads();

    //
    // Each thread now represents one expert
    //

    // Reduce histograms with atomics.
    if (threadIdx.x < params.mNumExperts)
    {
        int32_t const localExpertCount = smemExpertCount[threadIdx.x];
        atomicAdd(&params.mPtrExpertCounts[threadIdx.x], localExpertCount);
    }
}
#else
__global__ void routingIndicesHistogramKernel(KernelParams params)
{
    assert(false && "routingIndicesHistogramKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __launch_bounds__(NumThreadsHist) routingIndicesOffsetsKernel(KernelParams params)
{
    using TypeExpW = typename KernelParams::TypeExpW;
    using TypePacked = PackedScoreIdx<TypeExpW>;
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreadsHist];
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreadsHist];
    __shared__ int32_t __attribute((aligned(128))) smemExpertTileOffset[NumThreadsHist];
    // needed for the exclusive sum of token offsets
    using Scan = hipcub::BlockScan<int32_t, NumThreadsHist, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    static constexpr int MaxExpandedIdxPerThread = NumEltsPerOffsetTilePerThread;
    static constexpr int MaxExpandedIdxPerBlock = NumThreadsHist * MaxExpandedIdxPerThread;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

    uint32_t const expandedIdxSize = params.mNumTokens * NumTopExperts;
    uint32_t const numTiles = (expandedIdxSize + MaxExpandedIdxPerBlock - 1) / (MaxExpandedIdxPerBlock);

    // Wait on primary grid.
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    // The expert offsets are common to all tiles of all blocks.
    // Load the histogram, scan it and write offsets to shared memory.
    // Note: the scan is redundant in all CTAs. Would it make sense to use an intermediate kernel for
    // the scan, with PDL?

    //
    // Each thread represents one expert.
    //

    // Get total count for this expert.
    int32_t count = (threadIdx.x < params.mNumExperts) ? params.mPtrExpertCounts[threadIdx.x] : 0;

    // Compute the runtime config for projections
    // Weather or not an expert is local is taken into account when the histogram is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    if (threadIdx.x < params.mNumExperts)
    {
        // Get the padded offset associated with this expert
        const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);

        // Write expert offsets to shared
        smemExpertOffset[threadIdx.x] = offset;
    }

    // Sync to make expert offsets available to all threads.
    __syncthreads();

    // The first block writes out padded count
    if (blockIdx.x == 0 && warpIdx == NumWarpsHist - 1 && cute::elect_one_sync())
    {
        const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    if (threadIdx.x < params.mNumExperts)
    {
        // Strided loop to share this work between blocks.
        for (int32_t cta = blockIdx.x; cta < numCta; cta += gridDim.x)
        {
            const int32_t localExpertIdx
                = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
            params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
            params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta]
                = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
                    mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + count);
        }
    }

    //
    // Now loop on indices and compute offsets.
    //

    // Grid-stride loop on 1D "tiles" of input indices.
    for (uint32_t tileIdx = blockIdx.x; tileIdx < numTiles; tileIdx += gridDim.x)
    {
        if (tileIdx > 0)
        {
            // Sync for safe reuse of smem buffers.
            __syncthreads();
        }

        // Pre-fill the counts with 0
        if (threadIdx.x < params.mNumExperts)
        {
            smemExpertCount[threadIdx.x] = 0;
        }
        __syncthreads();

        // each thread keeps has some number of "expanded indexes" assigned to it
        // for each of these, we keep the associated expert and offset within expert in registers
        int32_t expertIndexes[MaxExpandedIdxPerThread];
        int32_t expertOffsets[MaxExpandedIdxPerThread];
        auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;

        // Define a lambda to avoid code duplication in branches.
        auto loopBody = [&](int ii, int expandedIdx)
        {
            TypePacked scoreIdx = params.mPtrExpertIdx[expandedIdx];
            expertIndexes[ii] = scoreIdx.idx;
            // check whether this expert is local to our GPU at all and ignore if not
            auto localExpertIdx = scoreIdx.idx - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + scoreIdx.idx, 1) : 0;
        };

        // For all tiles but the last, all indices are in bounds.
        if (tileIdx < numTiles - 1)
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                loopBody(ii, expandedIdx);
            }
        }
        else
        {
            // For the last tile, we need to exit the loop when out of bounds.
            // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
            // time, and branch between a fast path without bound checks and a slow path with bound checks
            int constexpr IterStride = 4;
            static_assert(MaxExpandedIdxPerThread % IterStride == 0);

#pragma unroll
            for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride)
            {
                // Whether it's safe to do multiple iterations without bound checks.
                bool const takeFastPath
                    = tileIdx * MaxExpandedIdxPerBlock + (ii0 + IterStride) * NumThreadsHist <= expandedIdxSize;
                if (takeFastPath)
                {
#pragma unroll
                    for (int32_t jj = 0; jj < IterStride; jj++)
                    {
                        int const ii = ii0 + jj;
                        auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                        loopBody(ii, expandedIdx);
                    }
                }
                else
                {
                    bool doBreak = false;
#pragma unroll
                    for (int32_t jj = 0; jj < IterStride; jj++)
                    {
                        int const ii = ii0 + jj;
                        auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                        if (expandedIdx >= expandedIdxSize)
                        {
                            doBreak = true;
                            break;
                        }
                        loopBody(ii, expandedIdx);
                    }
                    if (doBreak)
                    {
                        break;
                    }
                }
            }
        }

        // Make local histogram (token counts per expert) available to all threads in the block.
        __syncthreads();

        //
        // Each thread now represents one expert
        //

        if (threadIdx.x < params.mNumExperts)
        {
            // Add the local bin count to the common bin count and get a per-CTA offset. We use the second
            // half of the histogram buffer for this histogram, because the first half already holds the
            // reduced histogram from the previous kernel.
            int32_t const localExpertCount = smemExpertCount[threadIdx.x];
            int32_t const tileExpertOffset
                = atomicAdd(&params.mPtrExpertCounts[params.mNumExperts + threadIdx.x], localExpertCount);

            // Make per-expert tile offsets available to all threads in the block.
            smemExpertTileOffset[threadIdx.x] = tileExpertOffset + smemExpertOffset[threadIdx.x];
        }
        __syncthreads();

        // Add tile offset and element offset and write to global memory.
        auto storeLoopBody = [&](int ii, int expandedIdx)
        {
            int32_t expertIdx = expertIndexes[ii];
            // check whether this expert is local to our GPU at all
            auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            auto tokenIdx = expandedIdx / NumTopExperts;
            auto permutedIdx = isLocalExpert ? (expertOffsets[ii] + smemExpertTileOffset[expertIdx]) : int32_t{-1};
            if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
            {
                params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
            }
            if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
            {
                params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
            }
        };
        // Bound checks only in last tile.
        if (tileIdx < numTiles - 1)
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                storeLoopBody(ii, expandedIdx);
            }
        }
        else
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                if (expandedIdx >= expandedIdxSize)
                {
                    break;
                }
                storeLoopBody(ii, expandedIdx);
            }
        }
    }

// Trigger secondary kernel.
// Note: this does not guarantee the visibility of prior writes unless the consumer executes a
// dependency sync.
#if !defined(PDL_PROFILE) || PDL_PROFILE == 0
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
#endif
}
#else
__global__ void routingIndicesOffsetsKernel(KernelParams params)
{
    assert(false && "routingIndicesOffsetsKernel is only supported on SM90+ architectures");
}
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    TLLM_CHECK_WITH_INFO(data.mPtrExpertIdx != nullptr || data.mPtrScores != nullptr,
        "Routing kernel requires at least one input parameter");
    TLLM_CHECK_WITH_INFO(data.mPtrPermutedIdxSize != nullptr && data.mPtrCtaIdxXyToBatchIdx != nullptr
            && data.mPtrCtaIdxXyToMnLimit != nullptr && data.mPtrNumNonExitingCtas != nullptr,
        "Llama4 routing kernel expects permuted idx and grouped Gemm launch config buffers");
    TLLM_CHECK_WITH_INFO(
        data.mTopK == NumTopExperts, "Routing kernel expects %d topK experts (for now)", NumTopExperts);
    TLLM_CHECK_WITH_INFO(data.mNumExperts <= MaxNumExperts,
        "Routing kernel expects #experts %d to be at most max #experts %d", data.mNumExperts, MaxNumExperts);
    static_assert(MaxNumExperts <= NumThreads, "#experts must be bounded by #threads");
    static_assert(MaxNumExperts <= NumThreadsHist, "#experts must be bounded by #threads");
    TLLM_CHECK_WITH_INFO(
        data.mNumExperts % 4 == 0, "Routing kernel expects #experts %d to be a multiple of 4.", data.mNumExperts);
    TLLM_CHECK_WITH_INFO(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got %d", data.mPaddingLog2);

    if (data.mPtrPermutedIdxToTokenIdx != nullptr)
    {
        // need to set all values to -1 before running the kernel
        auto maxPermutedSize
            = data.mNumTokens * data.mTopK + (data.mNumExperts << data.mPaddingLog2) - data.mNumExperts;
        // note that a value of -1 per byte works for any size of signed integer
        // to set each full value to the logical value -1
        TLLM_CUDA_CHECK(hipMemsetAsync(data.mPtrPermutedIdxToTokenIdx, -1,
            static_cast<size_t>(maxPermutedSize) * sizeof(int32_t), (hipStream_t) stream));
    }

    bool const useSingleWarp = (data.mPtrScores == nullptr && data.mNumTokens <= WarpKernelMaxNumTokens)
        || data.mNumTokens < WarpKernelMaxNumTokens;
    bool const useSingleCluster
        = data.mNumTokens <= (data.mPtrScores != nullptr ? MaxNumTokensSingleClusterScores : MaxNumTokensSingleCluster);
    if (!useSingleCluster)
    {
        TLLM_CHECK_WITH_INFO(
            data.mPtrExpertIdx != nullptr, "When #tokens is large, `mPtrExpertIdx` is a required input.");
        TLLM_CHECK_WITH_INFO(
            data.mPtrExpertCounts != nullptr, "When #tokens is large, `mPtrExpertCounts` is a required input.");
        // Reset the global histograms (not used in single-cluster code path).
        TLLM_CUDA_CHECK(hipMemsetAsync(data.mPtrExpertCounts, 0,
            static_cast<size_t>(2 * data.mNumExperts) * sizeof(int32_t), (hipStream_t) stream));
    }

    if (useSingleWarp)
    {
        LAUNCH_EXPW_ONLY(data,
            /*coopLaunch=*/false, routingIndicesWarpKernel, 1, WarpSize,
            /*smemSize=*/0, // No dynamic smem
            stream);
    }
    else if (useSingleCluster)
    {
        LAUNCH_EXPW_ONLY(data,
            /*coopLaunch=*/false, routingIndicesClusterKernel, NumBlocksPerCluster, NumThreads,
            /*smemSize=*/0, // No dynamic smem
            stream);
    }
    else
    {
        const uint32_t expandedIdxSize = data.mNumTokens * NumTopExperts;

        const uint32_t histogramEltsPerBlock = 8 * NumThreadsHist;
        const uint32_t offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreadsHist;

        // Limit grid size (all kernels use a grid-stride loop).
        const uint32_t maxNumBlocks = 1024;

        int const numBlocksHistogram
            = std::min((expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
        int const numBlocksOffsets
            = std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

        if (data.mPtrScores != nullptr)
        {
            LAUNCH_EXPW_ONLY(data,
                /*coopLaunch=*/false, routingIndicesHistogramScoresKernel, maxNumBlocks, NumThreadsHist,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
        LAUNCH_EXPW_ONLY(data,
            /*coopLaunch=*/false, routingIndicesHistogramKernel, numBlocksHistogram, NumThreadsHist,
            /*smemSize=*/0, // No dynamic smem
            stream);
        LAUNCH_EXPW_ONLY(data,
            /*coopLaunch=*/false, routingIndicesOffsetsKernel, numBlocksOffsets, NumThreadsHist,
            /*smemSize=*/0, // No dynamic smem
            stream);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace routingLlama4

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace routingQwen3
{

////////////////////////////////////////////////////////////////////////////////////////////////////

namespace cg = cooperative_groups;

////////////////////////////////////////////////////////////////////////////////////////////////////

static constexpr int NumThreads = 1024;
static constexpr int NumThreadsHist = 256;
static constexpr int NumBlocksPerCluster = 8;
static constexpr int WarpSize = 32;
static constexpr int NumWarps = NumThreads / WarpSize;
static constexpr int NumWarpsHist = NumThreadsHist / WarpSize;
static constexpr int NumTopExperts = 8;
static constexpr int MaxNumExperts = 128;
static constexpr int MaxNumTokensSingleCluster = NumBlocksPerCluster * NumThreads;
static constexpr int MaxNumTokensSingleClusterScores = NumBlocksPerCluster * NumWarps;

// Performance tuning knob.
static constexpr int NumEltsPerOffsetTilePerThread = 8;

#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ == 1000 && defined(__CUDA_ARCH_FEAT_SM100_ALL))
#define TLLM_GEN_ENABLE_FAST_REDUX
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename TypeExpW_>
struct TopKRedType
{
    using TypeExpW = TypeExpW_;
    static_assert(std::is_same_v<TypeExpW, float> || std::is_same_v<TypeExpW, cutlass::bfloat16_t>,
        "Top K reduction only implemented for float and Bf16");
    using TypeCmp = std::conditional_t<sizeof(TypeExpW) >= 4, double, float>;
    static constexpr int64_t Mask64 = 0x000000000000FFFF;
    static constexpr int32_t Mask32 = 0x0000FFFF;

    TypeCmp compVal;

    static __host__ __device__ inline TypeCmp makeCmpVal(TypeExpW val, int32_t idx = 0)
    {
        auto cmpVal = TypeCmp{val};
        TypeCmp cmpValWithIdx;
        if constexpr (sizeof(TypeExpW) >= 4)
        {
            auto cmpValIdx64 = reinterpret_cast<int64_t&>(cmpVal) | (Mask64& int64_t{idx});
            cmpValWithIdx = reinterpret_cast<TypeCmp&>(cmpValIdx64);
        }
        else
        {
            auto cmpValIdx32 = reinterpret_cast<int32_t&>(cmpVal) | (Mask32 & idx);
            cmpValWithIdx = reinterpret_cast<TypeCmp&>(cmpValIdx32);
        }
        return cmpValWithIdx;
    }

    static __host__ __device__ inline void unpack(TypeExpW& val, int32_t& idx, TypeCmp cmp)
    {
        if constexpr (sizeof(TypeExpW) >= 4)
        {
            idx = static_cast<int32_t>(reinterpret_cast<int64_t&>(cmp) & Mask64);
            auto val64 = reinterpret_cast<int64_t&>(cmp) & ~Mask64;
            val = static_cast<float>(reinterpret_cast<double&>(val64));
        }
        else
        {
            idx = reinterpret_cast<int32_t&>(cmp) & Mask32;
            auto val32 = reinterpret_cast<int32_t&>(cmp) >> 16;
            val = TypeExpW::bitcast(reinterpret_cast<uint16_t&>(val32));
        }
    }

    __host__ __device__ TopKRedType() = default;

    __host__ __device__ TopKRedType(TypeExpW val, int32_t idx)
        : compVal(makeCmpVal(val, idx))
    {
    }

    __host__ __device__ operator TypeCmp() const noexcept
    {
        return compVal;
    }

    __device__ inline TypeCmp reduce(cg::thread_block_tile<WarpSize> const& warp)
    {
#if defined(TLLM_GEN_ENABLE_FAST_REDUX)
        static constexpr bool UseCg = false;
#else
        static constexpr bool UseCg = true;
#endif
        if constexpr (UseCg || sizeof(TypeExpW) >= 4)
        {
            return cg::reduce(warp, compVal, cg::greater<TypeCmp>{});
        }
        else
        {
            float result;
            asm("redux.sync.max.f32 %0, %1, 0xffffffff;\n" : "=f"(result) : "f"(compVal));
            return result;
        }
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K_, bool Enable_>
struct TopKIdx
{
    // by default, empty
};

template <int K_>
struct TopKIdx<K_, true>
{
    static constexpr int K = K_;
    int32_t val[K];
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K, typename Type>
__device__ void reduceTopK(cg::thread_block_tile<WarpSize> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type value, int32_t idx, Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WarpSize, "Top K must have K < WarpSize");
    using RedType = TopKRedType<Type>;
    RedType topK{value, idx};
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        topK = kk > 0 && packedMax == topK.compVal ? RedType{minValue, idx} : topK;
        // get the next largest value
        packedMax = topK.reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

#define TOPK_SWAP(I, J)                                                                                                \
    {                                                                                                                  \
        auto pairMin = min(topK[I].compVal, topK[J].compVal);                                                          \
        auto pairMax = max(topK[I].compVal, topK[J].compVal);                                                          \
        topK[I].compVal = pairMax;                                                                                     \
        topK[J].compVal = pairMin;                                                                                     \
    }

template <int K, typename Type, int N, bool IsSorted = false>
__device__ void reduceTopK(cg::thread_block_tile<WarpSize> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type (&value)[N], int32_t (&idx)[N], Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WarpSize, "Top K must have K < WarpSize");
    static_assert(N > 0, "Top K must have N > 1");
    // static_assert(N <= K, "Top K must have N < K");
    using RedType = TopKRedType<Type>;
    RedType topK[N];
#pragma unroll
    for (int nn = 0; nn < N; ++nn)
    {
        topK[nn] = RedType{value[nn], idx[nn]};
    }

    if constexpr (!IsSorted)
    {
        TOPK_SWAP(0, 2);
        TOPK_SWAP(1, 3);

        TOPK_SWAP(0, 1);
        TOPK_SWAP(2, 3);

        TOPK_SWAP(1, 2);
    }
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        bool update = kk > 0 && packedMax == topK[0].compVal;
#pragma unroll
        for (int nn = 0; nn < N; ++nn)
        {
            topK[nn] = update && nn == N - 1 ? RedType{minValue, idx[nn]} : update ? topK[nn + 1] : topK[nn];
        }
        // get the next largest value
        packedMax = topK[0].reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

#undef TOPK_SWAP

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T mulLog2(T a, T bLog2)
{
    return a << bLog2;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T divUpLog2(T a, T bLog2)
{
    return ((a + (1 << bLog2) - 1) >> bLog2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__host__ __device__ constexpr T divUpMulLog2(T a, T bLog2)
{
    return mulLog2<T>(divUpLog2<T>(a, bLog2), bLog2);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

__host__ __device__ constexpr int32_t getBits(int32_t value, int idx)
{
    int mask = idx == 0 ? 0x000000FF : idx == 1 ? 0x0000FF00 : idx == 2 ? 0x00FF0000 : 0xFF000000;
    return (value & mask) >> (idx * 8);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <bool IsZero = false>
__host__ __device__ constexpr void setBits(int32_t& value, int32_t newBits, int idx)
{
    if constexpr (!IsZero)
    {
        int mask = idx == 0 ? 0xFFFFFF00 : idx == 1 ? 0xFFFF00FF : idx == 2 ? 0xFF00FFFF : 0x00FFFFFF;
        value &= mask;
    }
    value |= (newBits << (idx * 8));
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename TypeExpW, int VecSize>
__device__ void calcSoftmax(cg::thread_block_tile<WarpSize> const& warp, TypeExpW (&scores)[VecSize])
{
    TypeExpW maxScore = TypeExpW{-INFINITY};
    TypeExpW sumScore = TypeExpW{0.f};

    // Get the max score for each token
    for (int i = 0; i < VecSize; ++i)
    {
        maxScore = scores[i] >= maxScore ? scores[i] : maxScore;
    }
    maxScore = cg::reduce(warp, maxScore, cg::greater<TypeExpW>());

    // Get the summation of scores for each token
#pragma unroll
    for (int i = 0; i < VecSize; ++i)
    {
        scores[i] = static_cast<TypeExpW>(exp(scores[i] - maxScore));
        sumScore += scores[i];
    }
    sumScore = cg::reduce(warp, sumScore, cg::plus<TypeExpW>());

    // Normalize the scores
#pragma unroll
    for (int i = 0; i < VecSize; ++i)
    {
        scores[i] = static_cast<TypeExpW>(scores[i] / sumScore);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename TypeExpW>
__device__ TypeExpW calcSoftmax(
    cg::thread_block_tile<WarpSize> const& warp, TypeExpW score, int32_t laneIdx, int32_t NumTopExperts)
{
    TypeExpW maxScore = TypeExpW{-INFINITY};
    if (laneIdx < NumTopExperts)
    {
        maxScore = score >= maxScore ? score : maxScore;
    }
    maxScore = cg::reduce(warp, maxScore, cg::greater<TypeExpW>());

    float sumScore = float{0.f};
    float newScore;
    // Get the summation of scores for each token
    if (laneIdx < NumTopExperts)
    {
        newScore = static_cast<float>(score) - static_cast<float>(maxScore);
        newScore = static_cast<float>(exp(newScore));
        sumScore += newScore;
    }
    sumScore = cg::reduce(warp, sumScore, cg::plus<float>());

    if (laneIdx < NumTopExperts)
    {
        score = static_cast<TypeExpW>(newScore / sumScore);
    }

    return score;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams, bool DoSoftmaxBeforeTopK = false>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
__global__ void __cluster_dims__(NumBlocksPerCluster, 1, 1) __launch_bounds__(NumThreads)
    routingIndicesClusterKernel(KernelParams params)
{
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreads];
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreads];
    // number of tokens/expanded idx is bounded by total number of warps
    using TypeExpW = typename KernelParams::TypeExpW;

    using BaseType = std::conditional_t<DoSoftmaxBeforeTopK, float, TypeExpW>;
    using TypePacked = PackedScoreIdx<BaseType>;

    __shared__ TypePacked __attribute((aligned(128))) smemPackedScoreIdx[NumWarps * NumTopExperts];
    // Needed for the exclusive sum of token offsets.
    // Note: the scan might include more bins than needed, with bin counts of 0 to pad
    using Scan = hipcub::BlockScan<int32_t, NumThreads, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    // Number of threads in the cluster.
    static constexpr int NumThreadsPerCluster = NumThreads * NumBlocksPerCluster;
    // same as max num tokens*num top experts
    static constexpr int MaxExpandedIdxPerThread
        = (MaxNumTokensSingleCluster * NumTopExperts + NumThreadsPerCluster - 1) / NumThreadsPerCluster;

    uint32_t const clusterBlockRank = blockIdx.x;
    uint32_t const clusterThreadIdx = NumThreads * clusterBlockRank + threadIdx.x;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);
    int32_t const laneIdx = cutlass::arch::LaneId();

    auto expandedIdxSize = params.mNumTokens * NumTopExperts;
    auto warpTokenIdx = clusterBlockRank * NumWarps + warpIdx;
    auto scoreOffset = warpTokenIdx * params.mNumExperts;
    bool validToken = warpTokenIdx < params.mNumTokens;

    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

    // pre-fill the counts with 0
    if (threadIdx.x < params.mNumExperts)
    {
        smemExpertCount[threadIdx.x] = 0;
    }
    __syncthreads();

    // then wait on primary grid
    if constexpr (KernelParams::UsePdl)
    {
        cudaGridDependencySynchronize();
    }

    // initialize the mPtrPermutedIdxToTokenIdx
    if (params.mPtrPermutedIdxToTokenIdx != nullptr)
    {
        int32_t permIdxToTokenIdxNum
            = (params.mNumTokens * NumTopExperts + (params.mNumExperts << params.mPaddingLog2) - params.mNumExperts);
        for (int32_t i = clusterThreadIdx; i < permIdxToTokenIdxNum; i += NumThreadsPerCluster)
        {
            params.mPtrPermutedIdxToTokenIdx[i] = -1;
        }
        // A cluster synchronization is performed prior to setting mPtrPermutedIdxToTokenIdx at the end of the kernel.
        // Don't need to use __threadfence() here.
    }

    if (params.mPtrScores != nullptr)
    {
        // in this case, each warp represents a token
        BaseType score[MaxNumExperts / WarpSize];
        int32_t idx[MaxNumExperts / WarpSize];

        BaseType warpTopKScore[NumTopExperts];
        int32_t warpTopKExpertIdx[NumTopExperts];

        BaseType minScore = BaseType{-INFINITY};
        if (validToken)
        {
            for (int i = 0; i < MaxNumExperts / WarpSize; i++)
            {
                auto expertIdx = i * WarpSize + laneIdx;
                auto newScore = expertIdx < params.mNumExperts
                    ? static_cast<BaseType>(params.mPtrScores[scoreOffset + expertIdx])
                    : minScore;
                score[i] = newScore;
                idx[i] = expertIdx;
            }

            if constexpr (DoSoftmaxBeforeTopK)
            {
                calcSoftmax(warp, score);
            }

            // Get the top-k scores and their corresponding expert indices
            reduceTopK(warp, warpTopKScore, warpTopKExpertIdx, score, idx, minScore);

            // Normalize the scores
            if constexpr (DoSoftmaxBeforeTopK)
            {
                float sum = float{1.f};
                if (params.mNormTopkProb)
                {
                    sum = static_cast<float>(laneIdx < NumTopExperts ? warpTopKScore[laneIdx] : 0);
                    sum = cg::reduce(warp, sum, cg::plus<float>());
                }
                if (laneIdx < NumTopExperts)
                {
                    warpTopKScore[laneIdx] = warpTopKScore[laneIdx] / sum;
                    smemPackedScoreIdx[warpIdx * NumTopExperts + laneIdx]
                        = TypePacked{warpTopKScore[laneIdx], static_cast<int16_t>(warpTopKExpertIdx[laneIdx])};
                }
            }
            else
            {
                auto score = calcSoftmax(
                    warp, laneIdx < NumTopExperts ? warpTopKScore[laneIdx] : minScore, laneIdx, NumTopExperts);
                if (laneIdx < NumTopExperts)
                {
                    warpTopKScore[laneIdx] = score;
                    smemPackedScoreIdx[warpIdx * NumTopExperts + laneIdx]
                        = TypePacked{warpTopKScore[laneIdx], static_cast<int16_t>(warpTopKExpertIdx[laneIdx])};
                }
            }
        } // end if (validToken)

        // make packed scores available to all threads in cluster
        __cluster_barrier_arrive();
        __cluster_barrier_wait();
    }

    // each thread keeps some number of "expanded indexes" assigned to it
    // note that expanded indexes simply represent tokens here.
    // for each of these, we keep the associated expert and offset within expert in registers
    int32_t expertIndexes[MaxExpandedIdxPerThread];
    int32_t expertOffsets[MaxExpandedIdxPerThread];
    auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;
    // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
    // time, and branch between a fast path without bound checks and a slow path with bound checks.
    // TODO(mjoux): potentially add this back for perf tuning
    // int constexpr IterStride = 4;
    // static_assert(MaxExpandedIdxPerThread % IterStride == 0);

    // Define a lambda to avoid code duplication in both branches.
    auto loopBody = [&](int ii, int expandedIdx)
    {
        TypePacked scoreIdx;
        if (params.mPtrScores != nullptr)
        {
            TypePacked const* remoteSmem
                = cg::cluster_group::map_shared_rank(smemPackedScoreIdx, expandedIdx / (NumWarps * NumTopExperts));
            scoreIdx = remoteSmem[expandedIdx % (NumWarps * NumTopExperts)];
        }
        else
        {
            scoreIdx = TypePacked{static_cast<BaseType>(params.mPtrExpertIdx[expandedIdx].score),
                static_cast<int16_t>(params.mPtrExpertIdx[expandedIdx].idx)};
        }
        expertIndexes[ii] = scoreIdx.idx;
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = scoreIdx.idx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + scoreIdx.idx, 1) : 0;
        if (params.mPtrExpertWeights != nullptr)
        {
            params.mPtrExpertWeights[expandedIdx] = static_cast<TypeExpW>(scoreIdx.score);
        }
    };

    if (clusterThreadIdx < expandedIdxSize)
    {
        loopBody(0, clusterThreadIdx);
    }

    // Make local histogram (token counts per expert) available to all threads in the cluster.
    __cluster_barrier_arrive();
    __cluster_barrier_wait();

    //
    // Each thread now represents one expert
    //

    // Total number of tokens for this expert.
    int32_t count = 0;
    // Per-expert offset for this block.
    int32_t blockExpertOffset = 0;

    if (threadIdx.x < params.mNumExperts)
    {
        // Get the histogram bin from each rank for this expert.
        int32_t expertCounts[NumBlocksPerCluster];
#pragma unroll
        for (int rank = 0; rank < NumBlocksPerCluster; rank++)
        {
            int32_t const* remoteSmem = cg::cluster_group::map_shared_rank(smemExpertCount, rank);
            expertCounts[rank] = rank * NumWarps < params.mNumTokens ? remoteSmem[threadIdx.x] : 0;
        }

        // Compute an exclusive prefix sum of the block-local count.
#pragma unroll
        for (int rank = 0; rank < NumBlocksPerCluster; rank++)
        {
            if (rank == clusterBlockRank)
            {
                blockExpertOffset = count;
            }
            count += expertCounts[rank];
        }
    }

    // Arrive: we do not access distributed shared memory after this point.
    __cluster_barrier_arrive();

    // Compute the runtime config for projections
    // Whether or not an expert is local is taken into account when smemExpertCount is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    if (threadIdx.x < params.mNumExperts)
    {
        // Strided loop to share this work between blocks.
        for (int32_t cta = clusterBlockRank; cta < numCta; cta += NumBlocksPerCluster)
        {
            const int32_t localExpertIdx
                = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
            params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
            params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta]
                = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
                    mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + count);
        }

        // get the padded offset associated with this expert
        const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);

        // write expert offsets to shared
        smemExpertOffset[threadIdx.x] = offset + blockExpertOffset;
    }

    // write out padded count
    if (clusterBlockRank == 0 && warpIdx == NumWarps - 1 && cute::elect_one_sync())
    {
        const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    // make expert offsets available to all threads
    __syncthreads();

    // Wait: we cannot exit while other blocks may be accessing the current block's shared memory.
    // Note: I observed a perf benefit to doing this before the final loop so the compiler can
    // implement break with EXIT.
    __cluster_barrier_wait();

    // trigger the secondary kernel when using PDL
    // We can't do it earlier because FC1 depends on the mPtrCtaIdxXyToBatchIdx,
    // mPtrCtaIdxXyToMnLimit, mPtrNumNonExitingCtas and mPtrTotalNumPaddedTokens
    // TODO: this is not sufficient to ensure visibility in the next kernel!
#if !defined(PDL_PROFILE) || PDL_PROFILE == 0
    if constexpr (KernelParams::UsePdl)
    {
        cudaTriggerProgrammaticLaunchCompletion();
    }
#endif

    // each thread has the same "expanded indexes" assigned to it as above
    // at this point, we know the final offsets of experts and the offsets within
    // experts, which allows writing the final index values

#pragma unroll
    for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ++ii)
    {
        auto expandedIdx = static_cast<int32_t>(clusterThreadIdx) + ii * NumThreadsPerCluster;
        if (expandedIdx >= expandedIdxSize)
        {
            break;
        }
        auto expertIdx = expertIndexes[ii];
        // check whether this expert is local to our GPU at all
        auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        auto tokenIdx = expandedIdx / NumTopExperts;
        auto permutedIdx = isLocalExpert ? int32_t{smemExpertOffset[expertIdx]} + expertOffsets[ii] : int32_t{-1};
        if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
        {
            params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
        }
        if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
        {
            params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
        }
    }
}
#else
__global__ void __launch_bounds__(NumThreads) routingIndicesClusterKernel(KernelParams /* params */)
{
    assert(false && "routingIndicesClusterKernel is only supported on SM90+ architectures");
}
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
////////////////////////////////////////////////////////////////////////////////////////////////////

// this kernel is needed in case we have scores as input for the histogram kernel
template <typename KernelParams, bool DoSoftmaxBeforeTopK = true>
__global__ void __launch_bounds__(NumThreadsHist) routingIndicesHistogramScoresKernel(KernelParams params)
{
    using TypeExpW = typename KernelParams::TypeExpW;

    using BaseType = std::conditional_t<DoSoftmaxBeforeTopK, float, TypeExpW>;

    static constexpr int VecSize = MaxNumExperts / WarpSize;
    // we assume that #experts is a multiple of 4, so VecSize must be 4.
    static_assert(VecSize == 4);

    int32_t const laneIdx = cutlass::arch::LaneId();
    int32_t const warpIdx = threadIdx.x / WarpSize;
    int32_t const globalWarpIdx = blockIdx.x * NumWarpsHist + warpIdx;
    int32_t const globalWarpStride = gridDim.x * NumWarpsHist;
    BaseType minScore = BaseType{-INFINITY};
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WarpSize>(block);

    // Wait on primary grid.
    if constexpr (KernelParams::UsePdl)
    {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        cudaGridDependencySynchronize();
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    }

    // initialize the mPtrPermutedIdxToTokenIdx
    int32_t globalThreadIdx = globalWarpIdx * WarpSize + laneIdx;
    int32_t globalThreadStride = globalWarpStride * WarpSize;
    if (params.mPtrPermutedIdxToTokenIdx != nullptr)
    {
        int32_t permIdxToTokenIdxNum
            = (params.mNumTokens * NumTopExperts + (params.mNumExperts << params.mPaddingLog2) - params.mNumExperts);
        for (int32_t i = globalThreadIdx; i < permIdxToTokenIdxNum; i += globalThreadStride)
        {
            params.mPtrPermutedIdxToTokenIdx[i] = -1;
        }
    }

    // initialize the mPtrExpertCounts
    if (params.mPtrExpertCounts != nullptr)
    {
        int32_t expertCountsNum = 2 * params.mNumExperts;
        for (int32_t i = globalThreadIdx; i < expertCountsNum; i += globalThreadStride)
        {
            params.mPtrExpertCounts[i] = 0;
        }
    }

    // Trigger secondary kernel.
    if constexpr (KernelParams::UsePdl)
    {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        cudaTriggerProgrammaticLaunchCompletion();
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    }

    // in this case, each warp represents a token, and we use a grid-stride loop
    // over all warps/tokens
    for (int tokenIdx = globalWarpIdx; tokenIdx < params.mNumTokens; tokenIdx += globalWarpStride)
    {
        auto scoreOffset = tokenIdx * params.mNumExperts;
        BaseType allScores[VecSize];
        int32_t allExpertIdx[VecSize];
        BaseType warpTopKScore[NumTopExperts];
        int32_t warpTopKExpertIdx[NumTopExperts];

        //@TODO：optimize this part with vectorized loading

#pragma unroll
        for (int i = 0; i < VecSize; ++i)
        {
            auto expertIdx = i * WarpSize + laneIdx;
            auto newScore = expertIdx < params.mNumExperts
                ? static_cast<BaseType>(params.mPtrScores[scoreOffset + expertIdx])
                : minScore;
            allScores[i] = newScore;
            allExpertIdx[i] = expertIdx;
        }

        if constexpr (DoSoftmaxBeforeTopK)
        {
            calcSoftmax(warp, allScores);
        }

        // Get the top-k scores and their corresponding expert indices
        reduceTopK(warp, warpTopKScore, warpTopKExpertIdx, allScores, allExpertIdx, minScore);
        __syncwarp(); //@TODO: check the synchronization

        // Normalize the scores
        if constexpr (DoSoftmaxBeforeTopK)
        {
            float sum = float{1.f};
            if (params.mNormTopkProb)
            {
                sum = static_cast<float>(laneIdx < NumTopExperts ? warpTopKScore[laneIdx] : 0);
                sum = cg::reduce(warp, sum, cg::plus<float>());
            }
            if (laneIdx < NumTopExperts)
            {
                warpTopKScore[laneIdx] = warpTopKScore[laneIdx] / sum;
            }
        }
        else
        {
            auto score = laneIdx < NumTopExperts ? warpTopKScore[laneIdx] : minScore;
            score = calcSoftmax(warp, score, laneIdx, NumTopExperts);
            if (laneIdx < NumTopExperts)
            {
                warpTopKScore[laneIdx] = score;
            }
        }
        for (int i = laneIdx; i < NumTopExperts; i += WarpSize)
        {
            PackedScoreIdx<TypeExpW> packedScore{
                static_cast<TypeExpW>(warpTopKScore[i]), static_cast<int16_t>(warpTopKExpertIdx[i])};
            params.mPtrExpertIdx[tokenIdx * NumTopExperts + i] = packedScore;
        }
    }
}

// Two-step approach (if number of tokens exceed limits of what cluster / cooperative launch
// variants can handle): in order to minimize the amount of data to exchange through global memory,
// we will compute the local histograms in smem twice: the first kernel will get us the total number
// of tokens per expert. The second kernel will use the smem and L2 atomics to get corresponding
// element and tile offsets.
//
// Note: the histogram calculation could also be fused with routingMainKernel, but this might be
// inefficient if we have one CTA per token doing a single global atomic.
template <typename KernelParams>
__global__ void __launch_bounds__(NumThreadsHist) routingIndicesHistogramKernel(KernelParams params)
{
    using TypeExpW = typename KernelParams::TypeExpW;

    using TypePacked = PackedScoreIdx<float>;
    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreadsHist];

    // For unrolling.
    uint32_t constexpr NumEltsPerThread = 8;

    // Pre-fill the counts with 0
    if (threadIdx.x < params.mNumExperts)
    {
        smemExpertCount[threadIdx.x] = 0;
    }
    __syncthreads();

    // Wait on primary grid and trigger secondary kernel.
    if constexpr (KernelParams::UsePdl)
    {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        cudaGridDependencySynchronize();
        cudaTriggerProgrammaticLaunchCompletion();
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    }

    uint32_t const expandedIdxSize = params.mNumTokens * NumTopExperts;
    uint32_t const localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;

    uint32_t const gridBlockOffset = blockIdx.x * NumThreadsHist;
    uint32_t const gridStride = gridDim.x * NumThreadsHist;

    // Define a lambda to avoid code duplication in branches.
    auto loopBody = [&](int expandedIdx)
    {
        PackedScoreIdx<TypeExpW> scoreIdx = params.mPtrExpertIdx[expandedIdx];
        // check whether this expert is local to our GPU at all and ignore if not
        auto localExpertIdx = scoreIdx.idx - params.mLocalExpertsStartIdx;
        auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
            && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
        if (isLocalExpert)
        {
            atomicAdd(&smemExpertCount[scoreIdx.idx], 1);
        }

        if (params.mPtrExpertWeights != nullptr)
        {
            params.mPtrExpertWeights[expandedIdx] = static_cast<TypeExpW>(scoreIdx.score);
        }
    };

    // Grid-stride loop.
    for (uint32_t expandedIdx0 = gridBlockOffset * NumEltsPerThread; expandedIdx0 < expandedIdxSize;
         expandedIdx0 += gridStride * NumEltsPerThread)
    {
        // Fast path if bound checks aren't necessary
        if (expandedIdx0 + NumEltsPerThread * NumThreadsHist <= expandedIdxSize)
        {
#pragma unroll
            for (uint32_t ii = 0; ii < NumEltsPerThread; ii++)
            {
                uint32_t expandedIdx = expandedIdx0 + ii * NumThreadsHist + threadIdx.x;
                loopBody(expandedIdx);
            }
        }
        else
        {
            for (uint32_t expandedIdx = expandedIdx0 + threadIdx.x; expandedIdx < expandedIdxSize;
                 expandedIdx += NumThreadsHist)
            {
                loopBody(expandedIdx);
            }
        }
    }
    __syncthreads();

    //
    // Each thread now represents one expert
    //

    // Reduce histograms with atomics.
    if (threadIdx.x < params.mNumExperts)
    {
        int32_t const localExpertCount = smemExpertCount[threadIdx.x];
        atomicAdd(&params.mPtrExpertCounts[threadIdx.x], localExpertCount);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename KernelParams>
__global__ void __launch_bounds__(NumThreadsHist) routingIndicesOffsetsKernel(KernelParams params)
{
    using TypeExpW = typename KernelParams::TypeExpW;
    using TypePacked = PackedScoreIdx<TypeExpW>;

    // number of experts is bounded by number of threads
    __shared__ int32_t __attribute((aligned(128))) smemExpertOffset[NumThreadsHist];
    __shared__ int32_t __attribute((aligned(128))) smemExpertCount[NumThreadsHist];
    __shared__ int32_t __attribute((aligned(128))) smemExpertTileOffset[NumThreadsHist];
    // needed for the exclusive sum of token offsets
    using Scan = hipcub::BlockScan<int32_t, NumThreadsHist, hipcub::BLOCK_SCAN_WARP_SCANS>;
    __shared__ typename Scan::TempStorage tempStorage;
    static constexpr int MaxExpandedIdxPerThread = NumEltsPerOffsetTilePerThread;
    static constexpr int MaxExpandedIdxPerBlock = NumThreadsHist * MaxExpandedIdxPerThread;

    int32_t const warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WarpSize, 0);

    uint32_t const expandedIdxSize = params.mNumTokens * NumTopExperts;
    uint32_t const numTiles = (expandedIdxSize + MaxExpandedIdxPerBlock - 1) / (MaxExpandedIdxPerBlock);

    // Wait on primary grid.
    if constexpr (KernelParams::UsePdl)
    {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        cudaGridDependencySynchronize();
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    }

    // The expert offsets are common to all tiles of all blocks.
    // Load the histogram, scan it and write offsets to shared memory.
    // Note: the scan is redundant in all CTAs. Would it make sense to use an intermediate kernel for
    // the scan, with PDL?

    //
    // Each thread represents one expert.
    //

    // Get total count for this expert.
    int32_t count = (threadIdx.x < params.mNumExperts) ? params.mPtrExpertCounts[threadIdx.x] : 0;

    // Compute the runtime config for projections
    // Whether or not an expert is local is taken into account when the histogram is computed
    // so we do not need to take it into account here.
    const int32_t numCta = divUpLog2<int32_t>(count, params.mPaddingLog2);
    int32_t ctaOffset;
    int32_t numNonExitingCtas;
    Scan(tempStorage).ExclusiveSum(numCta, ctaOffset, numNonExitingCtas);

    if (threadIdx.x < params.mNumExperts)
    {
        // Get the padded offset associated with this expert
        const int32_t offset = mulLog2<int32_t>(ctaOffset, params.mPaddingLog2);

        // Write expert offsets to shared
        smemExpertOffset[threadIdx.x] = offset;
    }

    // Sync to make expert offsets available to all threads.
    __syncthreads();

    // The first block writes out padded count
    if (blockIdx.x == 0 && warpIdx == NumWarpsHist - 1 && cute::elect_one_sync())
    {
        const int32_t permutedIdxSize = mulLog2<int32_t>(numNonExitingCtas, params.mPaddingLog2);
        params.mPtrPermutedIdxSize[0] = permutedIdxSize;
        params.mPtrNumNonExitingCtas[0] = numNonExitingCtas;
    }

    if (threadIdx.x < params.mNumExperts)
    {
        // Strided loop to share this work between blocks.
        for (int32_t cta = blockIdx.x; cta < numCta; cta += gridDim.x)
        {
            const int32_t localExpertIdx
                = (threadIdx.x - params.mLocalExpertsStartIdx) >> params.mLocalExpertsStrideLog2;
            params.mPtrCtaIdxXyToBatchIdx[ctaOffset + cta] = localExpertIdx;
            params.mPtrCtaIdxXyToMnLimit[ctaOffset + cta]
                = min(mulLog2<int32_t>(ctaOffset + cta + 1, params.mPaddingLog2),
                    mulLog2<int32_t>(ctaOffset, params.mPaddingLog2) + count);
        }
    }

    //
    // Now loop on indices and compute offsets.
    //

    // Grid-stride loop on 1D "tiles" of input indices.
    for (uint32_t tileIdx = blockIdx.x; tileIdx < numTiles; tileIdx += gridDim.x)
    {
        if (tileIdx > 0)
        {
            // Sync for safe reuse of smem buffers.
            __syncthreads();
        }

        // Pre-fill the counts with 0
        if (threadIdx.x < params.mNumExperts)
        {
            smemExpertCount[threadIdx.x] = 0;
        }
        __syncthreads();

        // each thread keeps has some number of "expanded indexes" assigned to it
        // for each of these, we keep the associated expert and offset within expert in registers
        int32_t expertIndexes[MaxExpandedIdxPerThread];
        int32_t expertOffsets[MaxExpandedIdxPerThread];
        auto localExpertExtent = params.mNumLocalExperts << params.mLocalExpertsStrideLog2;

        // Define a lambda to avoid code duplication in branches.
        auto loopBody = [&](int ii, int expandedIdx)
        {
            PackedScoreIdx<TypeExpW> scoreIdx = params.mPtrExpertIdx[expandedIdx];
            expertIndexes[ii] = scoreIdx.idx;
            // check whether this expert is local to our GPU at all and ignore if not
            auto localExpertIdx = scoreIdx.idx - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            expertOffsets[ii] = isLocalExpert ? atomicAdd(smemExpertCount + scoreIdx.idx, 1) : 0;
        };

        // For all tiles but the last, all indices are in bounds.
        if (tileIdx < numTiles - 1)
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                loopBody(ii, expandedIdx);
            }
        }
        else
        {
            // For the last tile, we need to exit the loop when out of bounds.
            // In order to avoid a serialization LDG-ATOMS-LDG-ATOMS-..., we skip multiple iterations at a
            // time, and branch between a fast path without bound checks and a slow path with bound checks
            int constexpr IterStride = 4;
            static_assert(MaxExpandedIdxPerThread % IterStride == 0);

#pragma unroll
            for (int32_t ii0 = 0; ii0 < MaxExpandedIdxPerThread; ii0 += IterStride)
            {
                // Whether it's safe to do multiple iterations without bound checks.
                bool const takeFastPath
                    = tileIdx * MaxExpandedIdxPerBlock + (ii0 + IterStride) * NumThreadsHist <= expandedIdxSize;
                if (takeFastPath)
                {
#pragma unroll
                    for (int32_t jj = 0; jj < IterStride; jj++)
                    {
                        int const ii = ii0 + jj;
                        auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                        loopBody(ii, expandedIdx);
                    }
                }
                else
                {
                    bool doBreak = false;
#pragma unroll
                    for (int32_t jj = 0; jj < IterStride; jj++)
                    {
                        int const ii = ii0 + jj;
                        auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                        if (expandedIdx >= expandedIdxSize)
                        {
                            doBreak = true;
                            break;
                        }
                        loopBody(ii, expandedIdx);
                    }
                    if (doBreak)
                    {
                        break;
                    }
                }
            }
        }

        // Make local histogram (token counts per expert) available to all threads in the block.
        __syncthreads();

        //
        // Each thread now represents one expert
        //

        if (threadIdx.x < params.mNumExperts)
        {
            // Add the local bin count to the common bin count and get a per-CTA offset. We use the second
            // half of the histogram buffer for this histogram, because the first half already holds the
            // reduced histogram from the previous kernel.
            int32_t const localExpertCount = smemExpertCount[threadIdx.x];
            int32_t const tileExpertOffset
                = atomicAdd(&params.mPtrExpertCounts[params.mNumExperts + threadIdx.x], localExpertCount);

            // Make per-expert tile offsets available to all threads in the block.
            smemExpertTileOffset[threadIdx.x] = tileExpertOffset + smemExpertOffset[threadIdx.x];
        }
        __syncthreads();

        // Add tile offset and element offset and write to global memory.
        auto storeLoopBody = [&](int ii, int expandedIdx)
        {
            int32_t expertIdx = expertIndexes[ii];
            // check whether this expert is local to our GPU at all
            auto localExpertIdx = static_cast<int32_t>(expertIdx) - params.mLocalExpertsStartIdx;
            auto isLocalExpert = localExpertIdx >= 0 && localExpertIdx < localExpertExtent
                && (localExpertIdx & params.mLocalExpertsStrideLog2) == 0;
            auto tokenIdx = expandedIdx / NumTopExperts;
            auto permutedIdx = isLocalExpert ? (expertOffsets[ii] + smemExpertTileOffset[expertIdx]) : int32_t{-1};
            if (params.mPtrExpandedIdxToPermutedIdx != nullptr)
            {
                params.mPtrExpandedIdxToPermutedIdx[expandedIdx] = permutedIdx;
            }
            if (params.mPtrPermutedIdxToTokenIdx != nullptr && isLocalExpert)
            {
                params.mPtrPermutedIdxToTokenIdx[permutedIdx] = tokenIdx;
            }
        };
        // Bound checks only in last tile.
        if (tileIdx < numTiles - 1)
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                storeLoopBody(ii, expandedIdx);
            }
        }
        else
        {
#pragma unroll
            for (int32_t ii = 0; ii < MaxExpandedIdxPerThread; ii += 1)
            {
                auto expandedIdx = tileIdx * MaxExpandedIdxPerBlock + ii * NumThreadsHist + threadIdx.x;
                if (expandedIdx >= expandedIdxSize)
                {
                    break;
                }
                storeLoopBody(ii, expandedIdx);
            }
        }
    }

// Trigger secondary kernel.
// Note: this does not guarantee the visibility of prior writes unless the consumer executes a
// dependency sync.
#if !defined(PDL_PROFILE) || PDL_PROFILE == 0
    if constexpr (KernelParams::UsePdl)
    {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        cudaTriggerProgrammaticLaunchCompletion();
#endif // if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    }
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void run(Data const& data, void* stream)
{
    TLLM_CHECK_WITH_INFO(data.mPtrExpertIdx != nullptr || data.mPtrScores != nullptr,
        "Routing kernel requires at least one input parameter");
    TLLM_CHECK_WITH_INFO(data.mPtrPermutedIdxSize != nullptr && data.mPtrCtaIdxXyToBatchIdx != nullptr
            && data.mPtrCtaIdxXyToMnLimit != nullptr && data.mPtrNumNonExitingCtas != nullptr,
        "Llama4 routing kernel expects permuted idx and grouped Gemm launch config buffers");
    TLLM_CHECK_WITH_INFO(
        data.mTopK == NumTopExperts, "Routing kernel expects %d topK experts (for now)", NumTopExperts);
    TLLM_CHECK_WITH_INFO(data.mNumExperts <= MaxNumExperts,
        "Routing kernel expects #experts %d to be at most max #experts %d", data.mNumExperts, MaxNumExperts);
    static_assert(MaxNumExperts <= NumThreads, "#experts must be bounded by #threads");
    static_assert(MaxNumExperts <= NumThreadsHist, "#experts must be bounded by #threads");
    TLLM_CHECK_WITH_INFO(
        data.mNumExperts % 4 == 0, "Routing kernel expects #experts %d to be a multiple of 4.", data.mNumExperts);
    TLLM_CHECK_WITH_INFO(data.mPaddingLog2 < 8, "Routing kernel expects padding log2 < 8, got %d", data.mPaddingLog2);

    bool const useSingleCluster
        = data.mNumTokens <= (data.mPtrScores != nullptr ? MaxNumTokensSingleClusterScores : MaxNumTokensSingleCluster);
    if (!useSingleCluster)
    {
        TLLM_CHECK_WITH_INFO(
            data.mPtrExpertIdx != nullptr, "When #tokens is large, `mPtrExpertIdx` is a required input.");
        TLLM_CHECK_WITH_INFO(
            data.mPtrExpertCounts != nullptr, "When #tokens is large, `mPtrExpertCounts` is a required input.");
    }

    if (useSingleCluster)
    {
        LAUNCH_EXPW_QWEN3(data, false, routingIndicesClusterKernel, NumBlocksPerCluster, NumThreads,
            /*smemSize=*/0, // No dynamic smem
            stream);
    }
    else
    {
        uint32_t const expandedIdxSize = data.mNumTokens * NumTopExperts;

        uint32_t const histogramEltsPerBlock = 8 * NumThreadsHist;
        uint32_t const offsetEltsPerBlock = NumEltsPerOffsetTilePerThread * NumThreadsHist;

        // Limit grid size (all kernels use a grid-stride loop).
        uint32_t const maxNumBlocks = 1024;

        int const numBlocksHistogram
            = std::min((expandedIdxSize + histogramEltsPerBlock - 1) / histogramEltsPerBlock, maxNumBlocks);
        int const numBlocksOffsets
            = std::min((expandedIdxSize + offsetEltsPerBlock - 1) / offsetEltsPerBlock, maxNumBlocks);

        if (data.mPtrScores != nullptr)
        {
            LAUNCH_EXPW_QWEN3(data, false, routingIndicesHistogramScoresKernel, maxNumBlocks, NumThreadsHist,
                /*smemSize=*/0, // No dynamic smem
                stream);
        }
        LAUNCH_EXPW_ONLY_QWEN3(data, false, routingIndicesHistogramKernel, numBlocksHistogram, NumThreadsHist,
            /*smemSize=*/0, // No dynamic smem
            stream);
        LAUNCH_EXPW_ONLY_QWEN3(data, false, routingIndicesOffsetsKernel, numBlocksOffsets, NumThreadsHist,
            /*smemSize=*/0, // No dynamic smem
            stream);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace routingQwen3

////////////////////////////////////////////////////////////////////////////////////////////////////

} // namespace moe::dev
