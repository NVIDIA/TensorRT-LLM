#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "customAllReduceKernels.h"
#include "tensorrt_llm/common/cudaBf16Fallbacks.cuh"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/customAllReduceUtils.h"
#include "tensorrt_llm/common/dataType.h"
#include "tensorrt_llm/common/envUtils.h"
#include <hip/hip_cooperative_groups.h>
#include <tuple>
#include <type_traits>

namespace tensorrt_llm::kernels
{

using tensorrt_llm::common::divUp;
using tensorrt_llm::common::roundUp;

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void st_flag_release(uint32_t const& flag, uint32_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b32 [%1], %0;" ::"r"(flag), "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ uint32_t ld_flag_acquire(uint32_t* flag_addr)
{
    uint32_t flag;
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b32 %0, [%1];" : "=r"(flag) : "l"(flag_addr));
#endif
    return flag;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
//
using PackedFloat = union
{
    int4 packed;
    float unpacked[4];
};

using PackedHalf = union
{
    int4 packed;
    half2 unpacked[4];
};

template <typename T>
struct PackedOn16Bytes
{
};

template <>
struct PackedOn16Bytes<float>
{
    using Type = PackedFloat;
};

template <>
struct PackedOn16Bytes<half>
{
    using Type = PackedHalf;
};

#ifdef ENABLE_BF16
using PackedBFloat16 = union
{
    int4 packed;
    __hip_bfloat162 unpacked[4];
};

template <>
struct PackedOn16Bytes<__hip_bfloat16>
{
    using Type = PackedBFloat16;
};

#endif

// add two 128b data
template <typename T>
inline __device__ int4 add128b(T& a, T& b)
{
    T c;
    c.unpacked[0] = a.unpacked[0] + b.unpacked[0];
    c.unpacked[1] = a.unpacked[1] + b.unpacked[1];
    c.unpacked[2] = a.unpacked[2] + b.unpacked[2];
    c.unpacked[3] = a.unpacked[3] + b.unpacked[3];
    return c.packed;
}

__inline__ __device__ void multi_gpu_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
    size_t const world_size, int const tidx, int const bidx)
{
    // After this function, at least one block in each GPU has reached the barrier
    if (tidx < world_size)
    {
        // we can think of signals having the shape [world_size, world_size]
        // Dimension 0 is the "listening" dimension, dimension 1 is "emitting" dimension

        // Block 0 broadcasts its flag (local_rank on emitting dimension) to all receivers
        size_t offset = (flag % 2) ? world_size : 0;

        if (bidx == 0)
        {
            st_flag_release(flag, signals[tidx] + offset + local_rank);
        }

        // All blocks check that corresponding block 0 on other GPUs have set the flag
        // No deadlock because block #0 is always the first block started
        uint32_t* peer_barrier_d = signals[local_rank] + offset + tidx;
        while (ld_flag_acquire(peer_barrier_d) != flag)
        {
        }
    }

    __syncthreads();
}

__inline__ __device__ void block_barrier(uint32_t** signals, uint32_t const flag, size_t const local_rank,
    size_t const world_size, int const tidx, int const bidx, int const grid_size)
{
    // After this function, the block of id == bidx of each GPU has reached the barrier
    if (tidx < world_size)
    {
        // we can think of signals having the shape [world_size, 2, num_blocks, world_size]
        // (+ an offset on dim 2 to account for flags used in multi_gpu_barrier)
        // Dimension 0 is the "listening" dimension, dimension 3 is "emitting" dimension

        // Block broadcast its flag (local_rank on emitting dimension) to all receivers
        uint32_t flag_block_offset = world_size + bidx * world_size;

        if (flag % 2 == 1)
        {
            flag_block_offset += (grid_size + 1) * world_size;
        }

        st_flag_release(flag, signals[tidx] + flag_block_offset + local_rank);

        // Blocks check that corresponding blocks on other GPUs have also set the flag
        uint32_t* peer_barrier_d = signals[local_rank] + flag_block_offset + tidx;

        while (ld_flag_acquire(peer_barrier_d) != flag)
        {
        }
    }

    __syncthreads();
}

namespace reduce_fusion
{

inline __device__ float warp_reduce_sum(float val)
{
    val += __shfl_xor_sync(~0, val, 16);
    val += __shfl_xor_sync(~0, val, 8);
    val += __shfl_xor_sync(~0, val, 4);
    val += __shfl_xor_sync(~0, val, 2);
    val += __shfl_xor_sync(~0, val, 1);
    return val;
}

inline __device__ float block_reduce_sum(float val)
{
    __shared__ float smem[details::kWarpSize];
    int lane_id = threadIdx.x % details::kWarpSize, warp_id = threadIdx.x / details::kWarpSize,
        warp_num = blockDim.x / details::kWarpSize;
    val = warp_reduce_sum(val);
    if (lane_id == 0)
    {
        smem[warp_id] = val;
    }
    __syncthreads();
    val = lane_id < warp_num ? smem[lane_id] : 0.f;
    val = warp_reduce_sum(val);
    return val;
}

template <typename T, typename PackedStruct>
inline __device__ float accumulate(float acc, PackedStruct& vec)
{
    static constexpr int kLoopNum = sizeof(PackedStruct) / sizeof(T);
#pragma unroll
    for (int i = 0; i < kLoopNum; ++i)
    {
        float v = static_cast<float>(reinterpret_cast<T*>(vec.unpacked)[i]);
        acc += v * v;
    }
    return acc;
}

template <typename T, bool Affine, typename PackedStruct>
inline __device__ int4 rms_norm(float denom, PackedStruct& vec, PackedStruct& weight)
{
    static constexpr int kLoopNum = sizeof(PackedStruct) / sizeof(T);
    PackedStruct ret;
#pragma unroll
    for (int i = 0; i < kLoopNum; ++i)
    {
        float v1 = static_cast<float>(reinterpret_cast<T*>(vec.unpacked)[i]);
        if constexpr (Affine)
        {
            float v2 = static_cast<float>(reinterpret_cast<T*>(weight.unpacked)[i]);
            reinterpret_cast<T*>(ret.unpacked)[i] = static_cast<T>(v1 * denom * v2);
        }
        else
        {
            reinterpret_cast<T*>(ret.unpacked)[i] = static_cast<T>(v1 * denom);
        }
    }
    return ret.packed;
}

template <typename T, bool Bias = false, bool Residual = false, bool Affine = false, bool UseSmem = false>
__global__ void rms_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    extern __shared__ uint8_t smem_ptr[];
    T* smem = reinterpret_cast<T*>(smem_ptr);

    int bid = blockIdx.x, tid = threadIdx.x;

    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    if constexpr (Residual)
    {
        residual_buffer += block_offset;
    }
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    PackedStruct inter_vec, weight_vec;
    float acc = 0.f;
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        inter_vec.packed = *reinterpret_cast<int4 const*>(intermediate_buffer + offset);
        if constexpr (Bias)
        {
            PackedStruct bias_vec;
            bias_vec.packed = *reinterpret_cast<int4 const*>(bias_buffer + offset);
            inter_vec.packed = add128b(inter_vec, bias_vec);
        }
        if constexpr (Residual)
        {
            PackedStruct residual_vec;
            residual_vec.packed = *reinterpret_cast<int4 const*>(residual_buffer + offset);
            inter_vec.packed = add128b(inter_vec, residual_vec);
            *reinterpret_cast<int4*>(intermediate_buffer + offset) = inter_vec.packed;
        }
        acc = accumulate<T>(acc, inter_vec);
        if constexpr (UseSmem)
        {
            *reinterpret_cast<int4*>(&smem[offset]) = inter_vec.packed;
        }
    }
    acc = block_reduce_sum(acc);
    float denom = rsqrtf(acc / params.fusion_params.hidden_size + params.fusion_params.eps);
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        if constexpr (UseSmem)
        {
            inter_vec.packed = *reinterpret_cast<int4 const*>(&smem[offset]);
        }
        if constexpr (Affine)
        {
            weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + offset);
        }
        inter_vec.packed = rms_norm<T, Affine>(denom, inter_vec, weight_vec);
        *reinterpret_cast<int4*>(&local_final_output_buffer[offset]) = inter_vec.packed;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, bool Bias = false, bool Residual = false, bool Affine = false>
__global__ void rms_pre_post_norm_kernel(AllReduceParams params) // for gemma2 pre residual + post residual norm
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    int bid = blockIdx.x, tid = threadIdx.x;

    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T const* weight_buffer_pre_residual_norm
        = reinterpret_cast<T const*>(params.fusion_params.weight_buffer_pre_residual_norm);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    if constexpr (Residual)
    {
        residual_buffer += block_offset;
    }
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    PackedStruct inter_vec, weight_vec, weight_vec_pre_residual_norm, bias_vec;
    float acc = 0.f;
    float acc_pre_residual_norm = 0.f;
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        inter_vec.packed = *reinterpret_cast<int4 const*>(intermediate_buffer + offset);
        if constexpr (Bias)
        {
            bias_vec.packed = *reinterpret_cast<int4 const*>(bias_buffer + offset);
        }

        if constexpr (Bias)
        {
            inter_vec.packed = add128b(inter_vec, bias_vec);
        }

        // pre-residual norm.
        acc_pre_residual_norm = accumulate<T>(acc_pre_residual_norm, inter_vec);
        acc_pre_residual_norm = block_reduce_sum(acc_pre_residual_norm);
        float denom_pre_residual_norm
            = rsqrtf(acc_pre_residual_norm / params.fusion_params.hidden_size + params.fusion_params.eps);

        if constexpr (Affine)
        {
            weight_vec_pre_residual_norm.packed
                = *reinterpret_cast<int4 const*>(weight_buffer_pre_residual_norm + thread_offset);
        }
        inter_vec.packed = rms_norm<T, Affine>(denom_pre_residual_norm, inter_vec, weight_vec_pre_residual_norm);

        if constexpr (Residual)
        {
            PackedStruct residual_vec;
            residual_vec.packed = *reinterpret_cast<int4 const*>(residual_buffer + offset);
            inter_vec.packed = add128b(inter_vec, residual_vec);
            *reinterpret_cast<int4*>(intermediate_buffer + offset) = inter_vec.packed;
        }
        acc = accumulate<T>(acc, inter_vec);
    }
    acc = block_reduce_sum(acc);
    float denom = rsqrtf(acc / params.fusion_params.hidden_size + params.fusion_params.eps);
    for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
    {
        if constexpr (Affine)
        {
            weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + offset);
        }
        inter_vec.packed = rms_norm<T, Affine>(denom, inter_vec, weight_vec);
        *reinterpret_cast<int4*>(&local_final_output_buffer[offset]) = inter_vec.packed;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, bool Bias = false, bool Residual = false, bool Affine = false>
void rms_norm_kernel_launcher(AllReduceParams& params, hipStream_t stream, AllReduceFusionOp fusionOp)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    TLLM_CHECK(params.fusion_params.hidden_size % kPackedSize == 0);
    if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM)
    {
        TLLM_CHECK(params.fusion_params.hidden_size <= 8192);
    }
    int need_threads = params.fusion_params.hidden_size / kPackedSize;
    int cta_size;
    if (need_threads <= details::kMaxCtaSize)
    {
        cta_size = (need_threads + details::kWarpSize - 1) / details::kWarpSize * details::kWarpSize;
    }
    else
    {
        cta_size = details::kMaxCtaSize;
    }
    int cta_num = params.elts_total / params.fusion_params.hidden_size;
    int smem_size = 0;
    if (cta_size * details::kBytesPerAccess / sizeof(T) < params.fusion_params.hidden_size)
    {
        smem_size = params.fusion_params.hidden_size * sizeof(T);
        if (tensorrt_llm::common::getEnvEnablePDL())
        {
            TLLM_LOG_DEBUG("Enable PDL in rms_norm_kernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = cta_num;
            kernelConfig.blockDim = cta_size;
            kernelConfig.dynamicSmemBytes = smem_size;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
            {
                TLLM_CUDA_CHECK(
                    cudaLaunchKernelEx(&kernelConfig, rms_norm_kernel<T, Bias, Residual, Affine, true>, params));
            }
            else
            { // AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM
                TLLM_CUDA_CHECK(
                    cudaLaunchKernelEx(&kernelConfig, rms_pre_post_norm_kernel<T, Bias, Residual, Affine>, params));
            }
        }
        else
        {
            if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
            {
                rms_norm_kernel<T, Bias, Residual, Affine, true><<<cta_num, cta_size, smem_size, stream>>>(params);
            }
            else
            { // AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM
                rms_pre_post_norm_kernel<T, Bias, Residual, Affine><<<cta_num, cta_size, smem_size, stream>>>(params);
            }
        }
    }
    else
    {
        if (tensorrt_llm::common::getEnvEnablePDL())
        {
            TLLM_LOG_DEBUG("Enable PDL in rms_norm_kernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = cta_num;
            kernelConfig.blockDim = cta_size;
            kernelConfig.dynamicSmemBytes = smem_size;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
            {
                TLLM_CUDA_CHECK(
                    cudaLaunchKernelEx(&kernelConfig, rms_norm_kernel<T, Bias, Residual, Affine, false>, params));
            }
            else
            { // AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM
                TLLM_CUDA_CHECK(
                    cudaLaunchKernelEx(&kernelConfig, rms_pre_post_norm_kernel<T, Bias, Residual, Affine>, params));
            }
        }
        else
        {
            if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
            {
                rms_norm_kernel<T, Bias, Residual, Affine, false><<<cta_num, cta_size, smem_size, stream>>>(params);
            }
            else
            { // AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM
                rms_pre_post_norm_kernel<T, Bias, Residual, Affine><<<cta_num, cta_size, smem_size, stream>>>(params);
            }
        }
    }
}

template <typename T>
struct NegZero128b
{
    static constexpr int v = static_cast<int>(0x80008000);
    static constexpr int4 value = {v, v, v, v};
};

template <>
struct NegZero128b<float>
{
    static constexpr int v = static_cast<int>(0x80000000);
    static constexpr int4 value = {v, v, v, v};
};

template <typename T>
__device__ static constexpr int4 NegZero128b_v = NegZero128b<T>::value;

template <typename T>
__device__ __forceinline__ bool is_neg_zero(T& v);

template <>
__device__ __forceinline__ bool is_neg_zero<float>(float& v)
{
    uint32_t bits = *reinterpret_cast<uint32_t*>(&v);
    return bits == 0x80000000;
}

template <>
__device__ __forceinline__ bool is_neg_zero<half>(half& v)
{
    uint16_t bits = *reinterpret_cast<uint16_t*>(&v);
    return bits == 0x8000;
}

template <>
__device__ __forceinline__ bool is_neg_zero<__hip_bfloat16>(__hip_bfloat16& v)
{
    uint16_t bits = *reinterpret_cast<uint16_t*>(&v);
    return bits == 0x8000;
}

template <typename ValType, typename VecType>
__device__ __forceinline__ VecType remove_neg_zero(VecType const& vec)
{
    static constexpr int kIter = sizeof(VecType) / sizeof(ValType);
    using ReadOnlyValType = std::add_const_t<ValType>;
    VecType ret;
#pragma unroll
    for (int i = 0; i < kIter; ++i)
    {
        auto val = reinterpret_cast<ReadOnlyValType*>(&vec)[i];
        reinterpret_cast<ValType*>(&ret)[i] = is_neg_zero(val) ? static_cast<ValType>(0.f) : val;
    }
    return ret;
}

template <typename ValType, typename VecType>
__device__ __forceinline__ bool has_neg_zero(VecType const& vec)
{
    static constexpr int kIter = sizeof(VecType) / sizeof(ValType);
    using ReadOnlyValType = std::add_const_t<ValType>;
#pragma unroll
    for (int i = 0; i < kIter; ++i)
    {
        auto val = reinterpret_cast<ReadOnlyValType*>(&vec)[i];
        if (is_neg_zero(val))
        {
            return true;
        }
    }
    return false;
}

template <typename ValType, typename VecType>
__device__ __forceinline__ bool all_neg_zero(VecType const& vec)
{
    static constexpr int kIter = sizeof(VecType) / sizeof(ValType);
    using ReadOnlyValType = std::add_const_t<ValType>;
#pragma unroll
    for (int i = 0; i < kIter; ++i)
    {
        auto val = reinterpret_cast<ReadOnlyValType*>(&vec)[i];
        if (!is_neg_zero(val))
        {
            return false;
        }
    }
    return true;
}

__device__ __forceinline__ void st_global_release(int4 const& val, int4* addr)
{
    asm volatile("st.release.global.sys.v4.b32 [%4], {%0, %1, %2, %3};" ::"r"(val.x), "r"(val.y), "r"(val.z),
        "r"(val.w), "l"(addr));
}

__device__ __forceinline__ int4 ld_global_acquire(int4* addr)
{
    int4 val;
    asm volatile("ld.acquire.global.sys.v4.b32 {%0, %1, %2, %3}, [%4];"
                 : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                 : "l"(addr));
    return val;
}

__device__ __forceinline__ void st_global_volatile(int4 const& val, int4* addr)
{
    asm volatile("st.volatile.global.v4.b32 [%4], {%0, %1, %2, %3};" ::"r"(val.x), "r"(val.y), "r"(val.z), "r"(val.w),
        "l"(addr));
}

__device__ __forceinline__ int4 ld_global_volatile(int4* addr)
{
    int4 val;
    asm volatile("ld.volatile.global.v4.b32 {%0, %1, %2, %3}, [%4];"
                 : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                 : "l"(addr));
    return val;
}

template <typename ValType>
__device__ __forceinline__ void set_neg_zero(int4* addr)
{
    st_global_volatile(NegZero128b_v<ValType>, addr);
}

template <typename T, int RanksPerNode, bool PushMode>
struct Reducer;

template <typename T, int RanksPerNode>
struct Reducer<T, RanksPerNode, true>
{
    static __device__ __forceinline__ int4 allreduce(AllReduceParams& params, int global_offset)
    {
        using PackedStruct = typename PackedOn16Bytes<T>::Type;
        int ping = params.barrier_flag % 3;
        int pong = (params.barrier_flag + 2) % 3;
        T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
        T* local_shared_buffer = reinterpret_cast<T*>(
            params.fusion_params.lamport_peer_comm_buffer_ptrs[params.local_rank + ping * MAX_RANKS_PER_NODE]);
        T* local_clean_buffer = reinterpret_cast<T*>(
            params.fusion_params.lamport_peer_comm_buffer_ptrs[params.local_rank + pong * MAX_RANKS_PER_NODE]);
        local_input_buffer += global_offset;
        local_shared_buffer += global_offset;
        local_clean_buffer += global_offset;
        T* buffers[RanksPerNode];
#pragma unroll
        for (int ii = 0; ii < RanksPerNode; ++ii)
        {
            int rank = (params.local_rank + ii) % RanksPerNode;
            buffers[ii] = reinterpret_cast<T*>(
                              params.fusion_params.lamport_peer_comm_buffer_ptrs[rank + ping * MAX_RANKS_PER_NODE])
                + global_offset + params.local_rank * params.elts_total;
        }
        PackedStruct sum_vec, val;
        val.packed = remove_neg_zero<T>(*reinterpret_cast<int4 const*>(local_input_buffer));
#pragma unroll
        for (int ii = 1; ii < RanksPerNode; ++ii)
        {
            st_global_volatile(val.packed, reinterpret_cast<int4*>(buffers[ii]));
        }
        sum_vec.packed = val.packed;
#pragma unroll
        for (int ii = 1; ii < RanksPerNode; ++ii)
        {
            int rank = (params.local_rank + ii) % RanksPerNode;
            set_neg_zero<T>(reinterpret_cast<int4*>(local_clean_buffer + rank * params.elts_total));
        }
        PackedStruct vals[RanksPerNode - 1];
        bool done = false;
        while (!done)
        {
            done = true;
#pragma unroll
            for (int ii = 1; ii < RanksPerNode; ++ii)
            {
                int rank = (params.local_rank + ii) % RanksPerNode;
                vals[ii - 1].packed
                    = ld_global_volatile(reinterpret_cast<int4*>(local_shared_buffer + rank * params.elts_total));
            }
#pragma unroll
            for (int ii = 0; ii < RanksPerNode - 1; ii++)
            {
                done &= !has_neg_zero<T>(vals[ii].packed);
            }
        }

#pragma unroll
        for (int ii = 1; ii < RanksPerNode; ++ii)
        {
            sum_vec.packed = add128b(sum_vec, vals[ii - 1]);
        }
        return sum_vec.packed;
    }
};

template <typename T, int RanksPerNode>
struct Reducer<T, RanksPerNode, false>
{
    static __device__ __forceinline__ int4 allreduce(AllReduceParams& params, int global_offset)
    {
        using PackedStruct = typename PackedOn16Bytes<T>::Type;
        int ping = params.barrier_flag % 3;
        int pong = (params.barrier_flag + 2) % 3;
        T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
        T* local_shared_buffer = reinterpret_cast<T*>(
            params.fusion_params.lamport_peer_comm_buffer_ptrs[params.local_rank + ping * MAX_RANKS_PER_NODE]);
        T* local_clean_buffer = reinterpret_cast<T*>(
            params.fusion_params.lamport_peer_comm_buffer_ptrs[params.local_rank + pong * MAX_RANKS_PER_NODE]);
        local_input_buffer += global_offset;
        local_shared_buffer += global_offset;
        local_clean_buffer += global_offset;
        T* buffers[RanksPerNode];
#pragma unroll
        for (int ii = 0; ii < RanksPerNode; ++ii)
        {
            int rank = (params.local_rank + ii) % RanksPerNode;
            buffers[ii] = reinterpret_cast<T*>(
                              params.fusion_params.lamport_peer_comm_buffer_ptrs[rank + ping * MAX_RANKS_PER_NODE])
                + global_offset;
        }
        PackedStruct sum_vec, val;
        val.packed = remove_neg_zero<T>(*reinterpret_cast<int4 const*>(local_input_buffer));
        st_global_volatile(val.packed, reinterpret_cast<int4*>(local_shared_buffer));
        sum_vec.packed = val.packed;
#pragma unroll
        for (int ii = 1; ii < RanksPerNode; ++ii)
        {
            do
            {
                val.packed = ld_global_volatile(reinterpret_cast<int4*>(buffers[ii]));
            } while (has_neg_zero<T>(val.packed));
            sum_vec.packed = add128b(sum_vec, val);
        }
        set_neg_zero<T>(reinterpret_cast<int4*>(local_clean_buffer));
        return sum_vec.packed;
    }
};

template <int ClusterSize, typename T, int RanksPerNode, bool Bias = false, bool Affine = false, bool PushMode = true>
static __global__ void lamport_style_one_shot_all_reduce_norm_kernel(AllReduceParams params)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    namespace cg = cooperative_groups;
    static_assert(RanksPerNode <= MAX_RANKS_PER_NODE);
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    cg::cluster_group cluster = cg::this_cluster();

    __shared__ float cluster_acc, cluster_acc_sum;

    int bid = blockIdx.x, tid = threadIdx.x;
    int cluster_id = bid / ClusterSize, cluster_block_rank = bid % ClusterSize;

    int token_id = cluster_id;
    int cluster_offset = token_id * params.fusion_params.hidden_size;
    int block_offset = cluster_block_rank * params.fusion_params.hidden_size / ClusterSize;
    int thread_offset = tid * kPackedSize;

    int inner_token_offset = block_offset + thread_offset;
    int global_offset = cluster_offset + inner_token_offset;

    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    local_final_output_buffer += global_offset;
    intermediate_buffer += global_offset;
    residual_buffer += global_offset;
    bias_buffer += inner_token_offset;
    weight_buffer += inner_token_offset;

    PackedStruct weight_vec, bias_vec, residual_vec;
    residual_vec.packed = *reinterpret_cast<int4 const*>(residual_buffer);
    if constexpr (Bias)
    {
        bias_vec.packed = *reinterpret_cast<int4 const*>(bias_buffer);
    }
    if constexpr (Affine)
    {
        weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer);
    }

    cudaGridDependencySynchronize();

    float acc = 0.f;
    PackedStruct sum_vec;
    sum_vec.packed = Reducer<T, RanksPerNode, PushMode>::allreduce(params, global_offset);

    if constexpr (Bias)
    {
        sum_vec.packed = add128b(sum_vec, bias_vec);
    }
    sum_vec.packed = add128b(sum_vec, residual_vec);
    *reinterpret_cast<int4*>(intermediate_buffer) = sum_vec.packed;
    acc = accumulate<T>(acc, sum_vec);
    acc = block_reduce_sum(acc);
    if (ClusterSize > 1)
    {
        if (threadIdx.x == 0)
        {
            cluster_acc = acc;
        }
        cluster.sync();
        if (threadIdx.x == 0)
        {
            acc = 0.f;
#pragma unroll
            for (int ii = 0; ii < ClusterSize; ++ii)
            {
                acc += *cluster.map_shared_rank(&cluster_acc, ii);
            }
            cluster_acc_sum = acc;
        }
        __syncthreads();
        acc = cluster_acc_sum;
        cluster.sync();
    }

    float denom = rsqrtf(acc / params.fusion_params.hidden_size + params.fusion_params.eps);
    sum_vec.packed = rms_norm<T, Affine>(denom, sum_vec, weight_vec);
    *reinterpret_cast<int4*>(local_final_output_buffer) = sum_vec.packed;

    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

int heuristic_min_warp_number(int tp_size, int hidden_size)
{
    if (hidden_size >= 4096)
    {
        return 4;
    }
    if (tp_size == 2)
    {
        return 32;
    }
    else
    {
        return 16;
    }
}

template <typename T, int RanksPerNode, bool Bias, bool Affine>
void lamport_style_one_shot_all_reduce_norm_kernel_launcher(AllReduceParams params, hipStream_t stream)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    TLLM_CHECK(params.fusion_params.hidden_size % kPackedSize == 0);
    int threads_per_token = params.fusion_params.hidden_size / kPackedSize;
    int warps_per_token = (threads_per_token + details::kWarpSize - 1) / details::kWarpSize;
    int token_num = params.elts_total / params.fusion_params.hidden_size;
    int warp_min_number = heuristic_min_warp_number(RanksPerNode, params.fusion_params.hidden_size);
    int cluster_size = std::min(((warps_per_token + warp_min_number - 1) / warp_min_number), details::kClusterMaxSize);
    int cta_size = warps_per_token / cluster_size * details::kWarpSize;
    TLLM_CHECK(cta_size <= details::kMaxCtaSize);
    int cta_num = token_num * cluster_size;
    hipLaunchConfig_t kernel_config = {0};
    kernel_config.gridDim = cta_num;
    kernel_config.blockDim = cta_size;
    kernel_config.dynamicSmemBytes = 0;
    kernel_config.stream = stream;

    hipLaunchAttribute attribute[2];
    attribute[0].id = cudaLaunchAttributeClusterDimension;
    attribute[0].val.clusterDim.x = cluster_size;
    attribute[0].val.clusterDim.y = 1;
    attribute[0].val.clusterDim.z = 1;
    kernel_config.attrs = attribute;
    kernel_config.numAttrs = 1;
    if (tensorrt_llm::common::getEnvEnablePDL())
    {
        attribute[1].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attribute[1].val.programmaticStreamSerializationAllowed = 1;
        kernel_config.numAttrs++;
    }
#define LAUNCH_LAMPORT_KERNEL(CLUSTER_SIZE)                                                                            \
    if (cluster_size == CLUSTER_SIZE)                                                                                  \
    {                                                                                                                  \
        TLLM_CUDA_CHECK(cudaLaunchKernelEx(&kernel_config,                                                             \
            lamport_style_one_shot_all_reduce_norm_kernel<CLUSTER_SIZE, T, RanksPerNode, Bias, Affine>, params));      \
        return;                                                                                                        \
    }
    LAUNCH_LAMPORT_KERNEL(1);
    LAUNCH_LAMPORT_KERNEL(2);
    LAUNCH_LAMPORT_KERNEL(3);
    LAUNCH_LAMPORT_KERNEL(4);
    LAUNCH_LAMPORT_KERNEL(5);
    LAUNCH_LAMPORT_KERNEL(6);
    LAUNCH_LAMPORT_KERNEL(7);
    LAUNCH_LAMPORT_KERNEL(8);
#undef LAUNCH_LAMPORT_KERNEL
}

template <typename T, int RanksPerNode, bool Bias = false, bool Affine = false, bool UseSmem = false>
static __global__ void __launch_bounds__(1024, 1) one_shot_all_reduce_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    extern __shared__ uint8_t smem_ptr[];
    T* smem = reinterpret_cast<T*>(smem_ptr);

    int bid = blockIdx.x, tid = threadIdx.x;
    int norm_num = params.elts_total / params.fusion_params.hidden_size;
    int norm_per_block = (norm_num + gridDim.x - 1) / gridDim.x;
    int norm_this_block = std::min(norm_per_block, norm_num - bid * norm_per_block);

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * norm_per_block * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    local_input_buffer += block_offset;
    residual_buffer += block_offset;
    local_shared_buffer += block_offset;
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

    T* buffers[RanksPerNode];
#pragma unroll
    for (int ii = 0; ii < RanksPerNode; ++ii)
    {
        int rank = (params.local_rank + ii) % RanksPerNode;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    for (int offset = thread_offset; offset < norm_this_block * params.fusion_params.hidden_size;
         offset += blockDim.x * kPackedSize)
    {
        *reinterpret_cast<int4*>(&local_shared_buffer[offset])
            = *reinterpret_cast<int4 const*>(&local_input_buffer[offset]);
    }
    block_barrier(
        params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RanksPerNode, tid, bid, gridDim.x);
    for (int norm_idx = 0; norm_idx < norm_this_block; ++norm_idx)
    {
        int norm_offset = norm_idx * params.fusion_params.hidden_size;
        float acc = 0.f;
        PackedStruct sum_vec, weight_vec, bias_vec, residual_vec;
        for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
        {
            PackedStruct vals[RanksPerNode];
            sum_vec.packed = {0, 0, 0, 0};
            if constexpr (Bias)
            {
                bias_vec.packed = *reinterpret_cast<int4 const*>(&bias_buffer[offset]);
            }
            residual_vec.packed = *reinterpret_cast<int4 const*>(&residual_buffer[norm_offset + offset]);
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][block_offset + norm_offset + offset]);
            }
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                sum_vec.packed = add128b(sum_vec, vals[ii]);
            }
            if constexpr (Bias)
            {
                sum_vec.packed = add128b(sum_vec, bias_vec);
            }
            sum_vec.packed = add128b(sum_vec, residual_vec);
            *reinterpret_cast<int4*>(&intermediate_buffer[norm_offset + offset]) = sum_vec.packed;
            acc = accumulate<T>(acc, sum_vec);
            if constexpr (UseSmem)
            {
                *reinterpret_cast<int4*>(&smem[offset]) = sum_vec.packed;
            }
        }
        acc = block_reduce_sum(acc);
        float denom = rsqrtf(acc / params.fusion_params.hidden_size + params.fusion_params.eps);
        for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
        {
            if constexpr (UseSmem)
            {
                sum_vec.packed = *reinterpret_cast<int4 const*>(&smem[offset]);
            }
            if constexpr (Affine)
            {
                weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + offset);
            }
            sum_vec.packed = rms_norm<T, Affine>(denom, sum_vec, weight_vec);
            *reinterpret_cast<int4*>(&local_final_output_buffer[norm_offset + offset]) = sum_vec.packed;
        }
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, int RanksPerNode, bool Bias = false, bool Affine = false>
static __global__ void __launch_bounds__(1024, 1) one_shot_prenorm_all_reduce_norm_kernel(AllReduceParams params)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    int bid = blockIdx.x, tid = threadIdx.x;
    int norm_num = params.elts_total / params.fusion_params.hidden_size;
    int norm_per_block = (norm_num + gridDim.x - 1) / gridDim.x;
    int norm_this_block = std::min(norm_per_block, norm_num - bid * norm_per_block);

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T const* bias_buffer = reinterpret_cast<T const*>(params.fusion_params.bias_buffer);
    T const* residual_buffer = reinterpret_cast<T const*>(params.fusion_params.residual_buffer);
    T const* weight_buffer = reinterpret_cast<T const*>(params.fusion_params.weight_buffer);
    T const* weight_buffer_pre_residual_norm
        = reinterpret_cast<T const*>(params.fusion_params.weight_buffer_pre_residual_norm);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_final_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);
    T* intermediate_buffer = reinterpret_cast<T*>(params.fusion_params.intermediate_buffer);

    int block_offset = bid * norm_per_block * params.fusion_params.hidden_size;
    int thread_offset = tid * kPackedSize;

    local_input_buffer += block_offset;
    residual_buffer += block_offset;
    local_shared_buffer += block_offset;
    local_final_output_buffer += block_offset;
    intermediate_buffer += block_offset;

    T* buffers[RanksPerNode];
#pragma unroll
    for (int ii = 0; ii < RanksPerNode; ++ii)
    {
        int rank = (params.local_rank + ii) % RanksPerNode;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    for (int offset = thread_offset; offset < norm_this_block * params.fusion_params.hidden_size;
         offset += blockDim.x * kPackedSize)
    {
        *reinterpret_cast<int4*>(&local_shared_buffer[offset])
            = *reinterpret_cast<int4 const*>(&local_input_buffer[offset]);
    }
    block_barrier(
        params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RanksPerNode, tid, bid, gridDim.x);
    for (int norm_idx = 0; norm_idx < norm_this_block; ++norm_idx)
    {
        int norm_offset = norm_idx * params.fusion_params.hidden_size;
        float acc = 0.f;
        float acc_pre_residual_norm = 0.f;
        PackedStruct sum_vec, weight_vec, bias_vec, residual_vec, weight_vec_pre_residual_norm;
        for (int offset = thread_offset; offset < params.fusion_params.hidden_size; offset += blockDim.x * kPackedSize)
        {
            PackedStruct vals[RanksPerNode];
            sum_vec.packed = {0, 0, 0, 0};
            if constexpr (Bias)
            {
                bias_vec.packed = *reinterpret_cast<int4 const*>(&bias_buffer[offset]);
            }
            residual_vec.packed = *reinterpret_cast<int4 const*>(&residual_buffer[norm_offset + offset]);
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][block_offset + norm_offset + offset]);
            }
#pragma unroll
            for (int ii = 0; ii < RanksPerNode; ++ii)
            {
                sum_vec.packed = add128b(sum_vec, vals[ii]);
            }

            if constexpr (Bias)
            {
                sum_vec.packed = add128b(sum_vec, bias_vec);
            }

            // norm1 is pre-residual norm.
            acc_pre_residual_norm = accumulate<T>(acc_pre_residual_norm, sum_vec);

            acc_pre_residual_norm = block_reduce_sum(acc_pre_residual_norm);

            float denom_pre_residual_norm
                = rsqrtf(acc_pre_residual_norm / params.fusion_params.hidden_size + params.fusion_params.eps);
            if constexpr (Affine)
            {
                weight_vec_pre_residual_norm.packed
                    = *reinterpret_cast<int4 const*>(weight_buffer_pre_residual_norm + thread_offset);
            }
            sum_vec.packed = rms_norm<T, Affine>(denom_pre_residual_norm, sum_vec, weight_vec_pre_residual_norm);

            sum_vec.packed = add128b(sum_vec, residual_vec);
            *reinterpret_cast<int4*>(&intermediate_buffer[norm_offset + offset]) = sum_vec.packed;
            acc = accumulate<T>(acc, sum_vec);
        }
        acc = block_reduce_sum(acc);
        float denom = rsqrtf(acc / params.fusion_params.hidden_size + params.fusion_params.eps);
        if constexpr (Affine)
        {
            weight_vec.packed = *reinterpret_cast<int4 const*>(weight_buffer + thread_offset);
        }
        sum_vec.packed = rms_norm<T, Affine>(denom, sum_vec, weight_vec);
        *reinterpret_cast<int4*>(&local_final_output_buffer[norm_offset + thread_offset]) = sum_vec.packed;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T>
bool is_lamport_supported(int token_num, int hidden_size)
{
    static char* disableLamportReduceNormFusionChar = std::getenv("DISABLE_LAMPORT_REDUCE_NORM_FUSION");
    bool disableLamportReduceNormFusion
        = (disableLamportReduceNormFusionChar != nullptr) || common::getEnvForceDeterministicAllReduce();
    if (disableLamportReduceNormFusion)
        return false;
    static int sm = tensorrt_llm::common::getSMVersion();
    if (sm < 90)
    {
        return false;
    }
    if (!std::is_same_v<T, half> && !std::is_same_v<T, __hip_bfloat16>)
    {
        return false;
    }
    if (token_num > details::kLamportTokenNumThreshold)
    {
        return false;
    }
    if (hidden_size < details::kLamportHiddenSizeThreshold)
    {
        return false;
    }
    return true;
}

bool is_lamport_supported(nvinfer1::DataType dataType, int token_num, int hidden_size)
{
    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: return is_lamport_supported<float>(token_num, hidden_size);
    case nvinfer1::DataType::kHALF: return is_lamport_supported<half>(token_num, hidden_size);
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: return is_lamport_supported<__hip_bfloat16>(token_num, hidden_size);
#endif
    default: return false;
    }
}

template <typename T, int RanksPerNode, bool Bias, bool Affine>
void one_shot_all_reduce_norm_kernel_launcher(AllReduceParams& params, hipStream_t stream, AllReduceFusionOp fusionOp)
{
    int token_num = params.elts_total / params.fusion_params.hidden_size;

    if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM)
    {
        TLLM_CHECK(params.fusion_params.hidden_size <= 8192);
    }

    if (is_lamport_supported<T>(token_num, params.fusion_params.hidden_size)
        && (fusionOp != AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM))
    {
        lamport_style_one_shot_all_reduce_norm_kernel_launcher<T, RanksPerNode, Bias, Affine>(params, stream);
    }
    else
    {
        static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
        TLLM_CHECK(params.fusion_params.hidden_size % kPackedSize == 0);
        int need_threads = params.fusion_params.hidden_size / kPackedSize;
        int cta_size;
        if (need_threads <= details::kMaxCtaSize)
        {
            cta_size = (need_threads + details::kWarpSize - 1) / details::kWarpSize * details::kWarpSize;
        }
        else
        {
            cta_size = details::kMaxCtaSize;
        }
        int norm_num = params.elts_total / params.fusion_params.hidden_size;
        int cta_num = std::min(norm_num, static_cast<int>(MAX_ALL_REDUCE_BLOCKS));
        int smem_size = 0;

        if (cta_size * kPackedSize < params.fusion_params.hidden_size)
        {
            smem_size = params.fusion_params.hidden_size * sizeof(T);
            if (tensorrt_llm::common::getEnvEnablePDL())
            {
                TLLM_LOG_DEBUG("Enable PDL in one_shot_all_reduce_norm_kernel");

                hipLaunchConfig_t kernelConfig = {0};
                kernelConfig.gridDim = cta_num;
                kernelConfig.blockDim = cta_size;
                kernelConfig.dynamicSmemBytes = smem_size;
                kernelConfig.stream = stream;

                hipLaunchAttribute attribute[1];
                attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
                attribute[0].val.programmaticStreamSerializationAllowed = 1;
                kernelConfig.attrs = attribute;
                kernelConfig.numAttrs = 1;
                if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
                {
                    TLLM_CUDA_CHECK(cudaLaunchKernelEx(
                        &kernelConfig, one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, true>, params));
                }
                else
                { // fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM
                    TLLM_CUDA_CHECK(cudaLaunchKernelEx(
                        &kernelConfig, one_shot_prenorm_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine>, params));
                }
            }
            else
            {
                if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
                {
                    one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, true>
                        <<<cta_num, cta_size, smem_size, stream>>>(params);
                }
                else
                {
                    one_shot_prenorm_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine>
                        <<<cta_num, cta_size, smem_size, stream>>>(params);
                }
            }
        }
        else
        {
            if (tensorrt_llm::common::getEnvEnablePDL())
            {
                hipLaunchConfig_t kernelConfig = {0};
                kernelConfig.gridDim = cta_num;
                kernelConfig.blockDim = cta_size;
                kernelConfig.dynamicSmemBytes = smem_size;
                kernelConfig.stream = stream;

                hipLaunchAttribute attribute[1];
                attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
                attribute[0].val.programmaticStreamSerializationAllowed = 1;
                kernelConfig.attrs = attribute;
                kernelConfig.numAttrs = 1;

                TLLM_LOG_DEBUG("Enable PDL in one_shot_all_reduce_norm_kernel");
                if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
                {
                    TLLM_CUDA_CHECK(cudaLaunchKernelEx(
                        &kernelConfig, one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, false>, params));
                }
                else
                { // fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM
                    TLLM_CUDA_CHECK(cudaLaunchKernelEx(
                        &kernelConfig, one_shot_prenorm_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine>, params));
                }
            }
            else
            {
                if (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM)
                {
                    one_shot_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine, false>
                        <<<cta_num, cta_size, smem_size, stream>>>(params);
                }
                else
                {
                    one_shot_prenorm_all_reduce_norm_kernel<T, RanksPerNode, Bias, Affine>
                        <<<cta_num, cta_size, smem_size, stream>>>(params);
                }
            }
        }
    }
}

template <typename T>
__global__ void lamport_initialize_kernel(T* buffer, size_t size)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;
    for (size_t offset = (blockIdx.x * blockDim.x + threadIdx.x) * kPackedSize; offset < size;
         offset += gridDim.x * blockDim.x * kPackedSize)
    {
        set_neg_zero<T>(reinterpret_cast<int4*>(&buffer[offset]));
    }
}

template <typename T>
void lamport_initialize_kernel_launcher(void* buffer, size_t size, hipStream_t stream)
{
    static constexpr int kPackedSize = details::kBytesPerAccess / sizeof(T);
    int block_size = 1024;
    int grid_size = (size + 1024 * kPackedSize - 1) / (1024 * kPackedSize);
    lamport_initialize_kernel<T><<<grid_size, block_size, 0, stream>>>(reinterpret_cast<T*>(buffer), size);
}
}; // namespace reduce_fusion

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true, bool PUSH_MODE = false>
static __global__ void oneShotAllReduceKernel(AllReduceParams params)
{
    // Suppose that two GPUs participate in the AR exchange, and we start four blocks.
    // The message is partitioned into chunks as detailed below:
    //               message
    //       |-------------------|
    // GPU 0 | B0 | B1 | B2 | B3 |
    // GPU 1 | B0 | B1 | B2 | B3 |
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 copies the chunk it  is responsible for, from local_input to shareable buffer
    // 2. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier)
    // 3. B0 on GPU 0 pull and sum the chunk from GPU 1, writes the result to local_output
    //
    // With COPY_INPUT == false, skip step 1. and use gpu_barrier instead of block barrier during step 2.
    // We only to know if the other GPU as arrived at the AR kernel, that would mean that data is ready
    //
    // With PUSH_MODE, we consider that the shared buffer is of size:
    // params.peer_comm_buffer_ptrs: [world_size, world_size, message_size]
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 push the chunk is it responsible for into all other GPUs:
    //    params.peer_comm_buffer_ptrs[:, local_gpu, B0 slice]
    // 2. block sync so the block is shared by other GPUs
    // 3. Reduce along second dimension params.peer_comm_buffer_ptrs[local_gpu, :, B0 slice]

    int const bidx = blockIdx.x;
    int const tidx = threadIdx.x;
    int const grid_size = gridDim.x;

    // The number of elements packed into one for comms
    static constexpr int PACKED_ELTS = 16 / sizeof(T);
    using PackedStruct = typename PackedOn16Bytes<T>::Type;

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

    // Start and end offsets of the thread
    size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
    size_t const chunk_end = std::min((bidx + 1) * params.elts_per_block, params.elts_total);

    T* buffers[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        // buffers[0] is always the local buffers. Helps load balancing reads.
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    if constexpr (PUSH_MODE || COPY_INPUT)
    {
        // Copy from local buffer to shareable buffer
        for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * PACKED_ELTS)
        {
            if constexpr (PUSH_MODE)
            {
#pragma unroll
                for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
                {
                    *reinterpret_cast<int4*>(&buffers[ii][params.local_rank * params.elts_total + iter_offset])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
                }
            }
            else
            {
                *reinterpret_cast<int4*>(&local_shared_buffer[iter_offset])
                    = *reinterpret_cast<int4 const*>(&local_input_buffer[iter_offset]);
            }
        }

        // wait for equivalent blocks of other GPUs to have copied data to their shareable buffer
        block_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);
    }
    else
    {
        // In the non-copy case, we assume that once the kernel has been started, data is ready to be consumed
        multi_gpu_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t iter_offset = chunk_start; iter_offset < chunk_end; iter_offset += blockDim.x * PACKED_ELTS)
    {
        // Iterate over the different ranks/devices on the node to load the values.
        PackedStruct vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if constexpr (PUSH_MODE)
            {
                vals[ii].packed
                    = *reinterpret_cast<int4 const*>(&buffers[params.local_rank][ii * params.elts_total + iter_offset]);
            }
            else
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][iter_offset]);
            }
        }

        // Sum the values from the different ranks.
        PackedStruct sums;
        sums.packed = {0, 0, 0, 0};
#pragma unroll
        for (int rank = 0; rank < RANKS_PER_NODE; ++rank)
        {
            // Always reduce from rank 0 to ensure stable reduce order.
            int ii = (rank + RANKS_PER_NODE - params.local_rank) % RANKS_PER_NODE;
            sums.packed = add128b(sums, vals[ii]);
        }
        // Store to the destination buffer.
        *reinterpret_cast<int4*>(&local_output_buffer[iter_offset]) = sums.packed;
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <typename T, int RANKS_PER_NODE, bool COPY_INPUT = true, bool PUSH_MODE = false, bool Bias = false,
    bool Residual = false>
static __global__ void __launch_bounds__(512, 1) twoShotAllReduceKernel(AllReduceParams params)
{
    // Suppose that two GPUs participate in the AR exchange, and we start two blocks.
    // The message is partitioned into chunks as detailed below:
    //               message
    //       |-------------------|
    //       |--GPU 0--|--GPU 1--| (GPU responsibility parts)
    // GPU 0 | B0 | B1 | B0 | B1 |
    // GPU 1 | B0 | B1 | B0 | B1 |
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 copies all chunks is it responsible for, from local_input to shareable buffer
    // 2. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier #0)
    // 3. B0 on GPU 0 gather and sum the B0 chunks from GPU 1, that are in the GPU 0 responsibility
    //    part (the first half of the message, see GPU responsibility row above)
    // 3bis. Likewise, B0 on GPU 1 copies and sum the chunks for GPU 0,
    //       where GPU 1 is responsible: the second half of the message.
    // 4. B0 on GPU 0 and B0 on GPU 1 wait for each other (block_barrier #1)
    // 5. B0 writes result to local_output. It gathers each chunk from its responsible GPU.
    //    For example, here it reads the first chunk from GPU 0 and second chunk from GPU 1.
    //
    // With COPY_INPUT == false, skip step 1. and use gpu_barrier instead of block barrier during step 2.
    // We only to know if the other GPU as arrived at the AR kernel, that would mean that data is ready
    // to be read.
    //
    // Note that compared to one-shot, one block (CTA) writes multiple input chunks and write multiple output chunks.
    // However, it's only responsible for the summation of a single chunk.
    //
    // With PUSH_MODE, we consider that the shared buffer is of size:
    // params.peer_comm_buffer_ptrs: [world_size, world_size, message_size / world_size]
    //
    // Here the step-by-step behavior of one block:
    // 1. B0 push the chunks is it responsible for into the corresponding GPUs:
    //    params.peer_comm_buffer_ptrs[target_gpu, local_gpu, current B0 slice]
    // 2. block sync so the blocks have been shared by other GPUs
    // 3. Reduce along second dimension params.peer_comm_buffer_ptrs[local_gpu, :, B0 slice]
    // 4. block barrier (corresponding blocks have finished reduction)
    // 5. pull and write on local buffer, by reading params.peer_comm_buffer_ptrs[:, 0, B0 slice] (reduction result is
    //    written at index 0 of 2nd dim)

    int const bidx = blockIdx.x;
    int const tidx = threadIdx.x;
    int const grid_size = gridDim.x;

    // The number of elements packed into one for comms
    static constexpr int PACKED_ELTS = 16 / sizeof(T);
    using PackedType = typename PackedOn16Bytes<T>::Type;

    T const* local_input_buffer = reinterpret_cast<T const*>(params.local_input_buffer_ptr);
    T* local_shared_buffer = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[params.local_rank]);
    T* local_output_buffer = reinterpret_cast<T*>(params.local_output_buffer_ptr);

    size_t const chunk_start = bidx * params.elts_per_block + tidx * PACKED_ELTS;
    size_t const chunk_end = min(chunk_start + params.elts_per_block, params.elts_per_rank);

    T* buffers[RANKS_PER_NODE];
    int ranks[RANKS_PER_NODE];
#pragma unroll
    for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        // A mapping of the ranks to scatter reads as much as possible
        int rank = (params.local_rank + ii) % RANKS_PER_NODE;
        ranks[ii] = rank;
        buffers[ii] = reinterpret_cast<T*>(params.peer_comm_buffer_ptrs[rank]);
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif

    if constexpr (PUSH_MODE || COPY_INPUT)
    {
        // Copy all blocks from local buffer to shareable buffer
        for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
        {
#pragma unroll
            for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
            {
                size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
                if (offset_rank >= params.elts_total)
                {
                    continue;
                }

                if constexpr (PUSH_MODE)
                {
                    *reinterpret_cast<int4*>(&buffers[ii][params.local_rank * params.elts_per_rank + local_offset])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
                }
                else
                {
                    *reinterpret_cast<int4*>(&local_shared_buffer[offset_rank])
                        = *reinterpret_cast<int4 const*>(&local_input_buffer[offset_rank]);
                }
            }
        }
        block_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);
    }
    else
    {
        // In the non-copy case, we assume that once the kernel has been started, data is ready to be consumed
        multi_gpu_barrier(
            params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    }

    // Each block accumulates the values from the different GPUs on the same node.
    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
    {
        size_t const responsible_block_offset = local_offset + params.rank_offset;

        // Iterate over the different ranks/devices on the node to load the values.
        PackedType vals[RANKS_PER_NODE];
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if constexpr (PUSH_MODE)
            {
                vals[ii].packed
                    = *reinterpret_cast<int4 const*>(&local_shared_buffer[ii * params.elts_per_rank + local_offset]);
            }
            else
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&buffers[ii][responsible_block_offset]);
            }
        }

        // Sum the values from the different ranks.
        PackedType sums;
        sums.packed = {0, 0, 0, 0};
#pragma unroll
        for (int rank = 0; rank < RANKS_PER_NODE; ++rank)
        {
            // Always reduce from rank 0 to ensure stable reduce order.
            int ii = (rank + RANKS_PER_NODE - params.local_rank) % RANKS_PER_NODE;
            sums.packed = add128b(sums, vals[ii]);
        }

        // Store to the local buffer.
        if constexpr (PUSH_MODE)
        {
            *reinterpret_cast<int4*>(&local_shared_buffer[local_offset]) = sums.packed;
        }
        else
        {
            *reinterpret_cast<int4*>(&local_shared_buffer[responsible_block_offset]) = sums.packed;
        }
    }

    block_barrier(
        params.peer_barrier_ptrs_out, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx, grid_size);

    // Gather all needed elts from other intra-node ranks
    for (size_t local_offset = chunk_start; local_offset < chunk_end; local_offset += blockDim.x * PACKED_ELTS)
    {
#pragma unroll
        for (int ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            // use round-robin gathering from other ranks
            size_t offset_rank = ranks[ii] * params.elts_per_rank + local_offset;
            if (offset_rank >= params.elts_total)
            {
                continue;
            }
            PackedType sums, residual_vec, bias_vec;
            if constexpr (Bias)
            {
                bias_vec.packed
                    = *reinterpret_cast<int4 const*>(reinterpret_cast<T const*>(params.fusion_params.bias_buffer)
                        + offset_rank % params.fusion_params.hidden_size);
            }
            if constexpr (Residual)
            {
                residual_vec.packed = *reinterpret_cast<int4 const*>(
                    reinterpret_cast<T const*>(params.fusion_params.residual_buffer) + offset_rank);
            }
            if constexpr (PUSH_MODE)
            {
                *reinterpret_cast<int4*>(&local_output_buffer[offset_rank])
                    = *reinterpret_cast<int4*>(&buffers[ii][local_offset]);
                sums.packed = *reinterpret_cast<int4*>(&buffers[ii][local_offset]);
            }
            else
            {
                *reinterpret_cast<int4*>(&local_output_buffer[offset_rank])
                    = *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
                sums.packed = *reinterpret_cast<int4*>(&buffers[ii][offset_rank]);
            }
            if constexpr (Bias)
            {
                sums.packed = add128b(sums, bias_vec);
            }
            if constexpr (Residual)
            {
                sums.packed = add128b(sums, residual_vec);
            }
            *reinterpret_cast<int4*>(&local_output_buffer[offset_rank]) = sums.packed;
        }
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

bool configurationSupported(AllReduceStrategyType algo, size_t msg_size, size_t n_ranks, nvinfer1::DataType type)
{
    size_t elts_per_thread = 16 / common::getDTypeSize(type);
    int const msg_align = (algo == AllReduceStrategyType::TWOSHOT) ? n_ranks * elts_per_thread : elts_per_thread;
    bool supported_algo = (algo == AllReduceStrategyType::ONESHOT || algo == AllReduceStrategyType::TWOSHOT);
    return supported_algo && (msg_size % msg_align == 0);
}

std::tuple<int, int> kernelLaunchConfig(AllReduceStrategyType algo, AllReduceParams& params, size_t elts_per_thread)
{
    int blocks_per_grid = 1, threads_per_block = DEFAULT_BLOCK_SIZE;

    switch (algo)
    {
    case AllReduceStrategyType::ONESHOT:
    {
        TLLM_CHECK(params.elts_total % elts_per_thread == 0);
        size_t const total_threads = roundUp(params.elts_total / elts_per_thread, WARP_SIZE);
        threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
        blocks_per_grid = std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
        params.elts_per_block = roundUp(divUp(params.elts_total, blocks_per_grid), elts_per_thread);
        break;
    }
    case AllReduceStrategyType::TWOSHOT:
    {
        TLLM_CHECK(params.elts_total % (elts_per_thread * params.ranks_per_node) == 0);
        size_t const total_threads = roundUp(params.elts_total / (elts_per_thread * params.ranks_per_node), WARP_SIZE);

        /*
        threads_per_block = std::min(DEFAULT_BLOCK_SIZE, total_threads);
        blocks_per_grid = std::min(static_cast<size_t>(MAX_ALL_REDUCE_BLOCKS), divUp(total_threads, threads_per_block));
        */
        while (total_threads % blocks_per_grid != 0 || total_threads / blocks_per_grid > DEFAULT_BLOCK_SIZE)
        {
            blocks_per_grid += 1;
        }

        threads_per_block = total_threads / blocks_per_grid;

        // NOTE: need to adjust here
        if (blocks_per_grid > MAX_ALL_REDUCE_BLOCKS)
        {
            size_t iter_factor = 1;
            while (blocks_per_grid / iter_factor > MAX_ALL_REDUCE_BLOCKS || blocks_per_grid % iter_factor)
            {
                iter_factor += 1;
            }
            blocks_per_grid /= iter_factor;
        }
        params.elts_per_rank = params.elts_total / params.ranks_per_node;
        params.rank_offset = params.local_rank * params.elts_per_rank;
        params.elts_per_block = roundUp(divUp(params.elts_per_rank, blocks_per_grid), elts_per_thread);
        break;
    }
    default: TLLM_THROW("Algorithm not supported here.");
    }

    return std::make_tuple(blocks_per_grid, threads_per_block);
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false, bool Bias = false,
    bool Affine = false>
void AllReduceNormKernelLaunch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    TLLM_CHECK_WITH_INFO(
        (fusionOp == AllReduceFusionOp::RESIDUAL_RMS_NORM || fusionOp == AllReduceFusionOp::RESIDUAL_RMS_PREPOST_NORM),
        "Unsupported AllReduceFusionOp: %d", static_cast<int>(fusionOp));
    if (algo == AllReduceStrategyType::ONESHOT)
    {
        reduce_fusion::one_shot_all_reduce_norm_kernel_launcher<T, RANKS_PER_NODE, Bias, Affine>(
            params, stream, fusionOp);
    }
    else
    {
        TLLM_CHECK_WITH_INFO(!(USE_MEMCPY && PUSH_MODE), "Memcpy cannot be used with PUSH_MODE.");
        size_t elts_per_thread = 16 / sizeof(T);
        auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(algo, params, elts_per_thread);
        if (USE_MEMCPY)
        {
            hipMemcpyAsync(params.peer_comm_buffer_ptrs[params.local_rank], params.local_input_buffer_ptr,
                params.elts_total * sizeof(T), hipMemcpyDeviceToDevice, stream);
        }
        auto output_ptr = params.local_output_buffer_ptr;
        params.local_output_buffer_ptr = params.fusion_params.intermediate_buffer;

        if (tensorrt_llm::common::getEnvEnablePDL())
        {
            TLLM_LOG_DEBUG("Enable PDL in twoShotAllReduceKernel");
            hipLaunchConfig_t kernelConfig = {0};
            kernelConfig.gridDim = blocks_per_grid;
            kernelConfig.blockDim = threads_per_block;
            kernelConfig.dynamicSmemBytes = 0;
            kernelConfig.stream = stream;

            hipLaunchAttribute attribute[1];
            attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
            attribute[0].val.programmaticStreamSerializationAllowed = 1;
            kernelConfig.attrs = attribute;
            kernelConfig.numAttrs = 1;

            TLLM_CUDA_CHECK(cudaLaunchKernelEx(
                &kernelConfig, twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE, Bias, true>, params));
        }
        else
        {
            twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE, Bias, true>
                <<<blocks_per_grid, threads_per_block, 0, stream>>>(params);
        }
        params.local_output_buffer_ptr = output_ptr;
        reduce_fusion::rms_norm_kernel_launcher<T, false, false, Affine>(params, stream, fusionOp);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceNormDispatch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, true, true>(
            algo, config, fusionOp, params, stream);
    }
    else if (params.fusion_params.bias_buffer && !params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, true, false>(
            algo, config, fusionOp, params, stream);
    }
    else if (!params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, false, true>(
            algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceNormKernelLaunch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY, false, false>(
            algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceDispatch(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    TLLM_CHECK(fusionOp == AllReduceFusionOp::NONE);
    TLLM_CHECK_WITH_INFO(!(USE_MEMCPY && PUSH_MODE), "Memcpy cannot be used with PUSH_MODE.");
    size_t elts_per_thread = 16 / sizeof(T);
    auto [blocks_per_grid, threads_per_block] = kernelLaunchConfig(algo, params, elts_per_thread);
    if (USE_MEMCPY)
    {
        hipMemcpyAsync(params.peer_comm_buffer_ptrs[params.local_rank], params.local_input_buffer_ptr,
            params.elts_total * sizeof(T), hipMemcpyDeviceToDevice, stream);
    }
    if (algo == AllReduceStrategyType::ONESHOT)
    {
        auto* kernel_instance = &oneShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE>;
        hipLaunchConfig_t config;
        config.gridDim = blocks_per_grid;
        config.blockDim = threads_per_block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attribute[1];
        attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attribute[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.attrs = attribute;
        config.numAttrs = 1;
        cudaLaunchKernelEx(&config, kernel_instance, params);
    }
    else
    {
        auto* kernel_instance = &twoShotAllReduceKernel<T, RANKS_PER_NODE, !USE_MEMCPY, PUSH_MODE>;
        hipLaunchConfig_t config;
        config.gridDim = blocks_per_grid;
        config.blockDim = threads_per_block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attribute[1];
        attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attribute[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.attrs = attribute;
        config.numAttrs = 1;
        cudaLaunchKernelEx(&config, kernel_instance, params);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false, bool USE_MEMCPY = false>
void AllReduceDispatchMemcpy(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (fusionOp == AllReduceFusionOp::NONE)
    {
        TLLM_LOG_DEBUG("AllReduceDispatch enabled");
        AllReduceDispatch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY>(algo, config, fusionOp, params, stream);
    }
    else
    {
        TLLM_LOG_DEBUG("AllReduceNormDispatch enabled");
        AllReduceNormDispatch<T, RANKS_PER_NODE, PUSH_MODE, USE_MEMCPY>(algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE, bool PUSH_MODE = false>
void AllReduceDispatchPushMode(AllReduceStrategyType algo, AllReduceStrategyConfig config, AllReduceFusionOp fusionOp,
    AllReduceParams& params, hipStream_t stream)
{
    if (static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(config)
        & static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(AllReduceStrategyConfig::USE_MEMCPY))
    {
        AllReduceDispatchMemcpy<T, RANKS_PER_NODE, PUSH_MODE, true>(algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceDispatchMemcpy<T, RANKS_PER_NODE, PUSH_MODE, false>(algo, config, fusionOp, params, stream);
    }
}

template <typename T, int RANKS_PER_NODE> //, bool USE_MEMCPY = false, bool PUSH_MODE = false>
void AllReduceDispatchRanksPerNode(AllReduceStrategyType algo, AllReduceStrategyConfig config,
    AllReduceFusionOp fusionOp, AllReduceParams& params, hipStream_t stream)
{
    if (static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(config)
        & static_cast<std::underlying_type_t<AllReduceStrategyConfig>>(AllReduceStrategyConfig::PUSH_MODE))
    {
        AllReduceDispatchPushMode<T, RANKS_PER_NODE, true>(algo, config, fusionOp, params, stream);
    }
    else
    {
        AllReduceDispatchPushMode<T, RANKS_PER_NODE, false>(algo, config, fusionOp, params, stream);
    }
}

template <typename T>
void AllReduceDispatchType(AllReduceParams& params, AllReduceStrategyType strat, AllReduceStrategyConfig config,
    AllReduceFusionOp fusionOp, hipStream_t stream)
{
    switch (params.ranks_per_node)
    {
    case 2: AllReduceDispatchRanksPerNode<T, 2>(strat, config, fusionOp, params, stream); break;
    case 4: AllReduceDispatchRanksPerNode<T, 4>(strat, config, fusionOp, params, stream); break;
    case 6: AllReduceDispatchRanksPerNode<T, 6>(strat, config, fusionOp, params, stream); break;
    case 8: AllReduceDispatchRanksPerNode<T, 8>(strat, config, fusionOp, params, stream); break;
    case 16: AllReduceDispatchRanksPerNode<T, 16>(strat, config, fusionOp, params, stream); break;
    default: TLLM_THROW("Custom all reduce only supported on {2, 4, 6, 8, 16} GPUs per node.");
    }
}

AllReduceParams AllReduceParams::deserialize(int64_t* buffer, size_t tpSize, size_t tpRank, nvinfer1::DataType dataType,
    int token_num, int hidden_size, AllReduceFusionOp op)
{
    void* const* buffer_ptrs = reinterpret_cast<void* const*>(buffer);
    int flag_offset;
    if (op == AllReduceFusionOp::RESIDUAL_RMS_NORM
        && reduce_fusion::is_lamport_supported(dataType, token_num, hidden_size))
    {
        flag_offset = 0;
    }
    else
    {
        flag_offset = 1;
    }
    auto const flag_ptr
        = &buffer[tensorrt_llm::utils::customAllReduceUtils::NUM_POINTERS_PER_RANK * tpSize + flag_offset];
    // cannot use 0 since 0 represents released state for barrier
    *flag_ptr += 1;
    TLLM_LOG_TRACE("AllReduceParams's flag value is %d, flag offset %d", *flag_ptr, flag_offset);
    uint32_t flag_value = *flag_ptr;
    AllReduceParams params;
    // Even plugins use ping buffers, odd plugins use pong.
    // That way, we don't need to wait for other GPUs to be done
    // before copying input tensor to workspace.
    auto const buffer_offset = (flag_value % 2 == 0) ? 0 : tpSize;

    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_comm_buffer_ptrs[i] = buffer_ptrs[buffer_offset + i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_in[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[2 * tpSize + i]);
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_out[i] = reinterpret_cast<uint32_t*>(buffer_ptrs[3 * tpSize + i]);
    }
    params.barrier_flag = flag_value;
    params.ranks_per_node = tpSize;
    params.local_rank = tpRank;

    return params;
}

void customAllReduce(kernels::AllReduceParams& params, nvinfer1::DataType dataType, AllReduceStrategyType strat,
    AllReduceStrategyConfig config, AllReduceFusionOp fusionOp, hipStream_t stream)
{
    TLLM_CHECK_WITH_INFO(configurationSupported(strat, params.elts_total, params.ranks_per_node, dataType),
        "Custom all-reduce configuration unsupported");

    sync_check_cuda_error(stream);

    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: AllReduceDispatchType<float>(params, strat, config, fusionOp, stream); break;
    case nvinfer1::DataType::kHALF: AllReduceDispatchType<half>(params, strat, config, fusionOp, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        AllReduceDispatchType<__hip_bfloat16>(params, strat, config, fusionOp, stream);
        break;
#endif
    default: TLLM_THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error(stream);
}

template <typename T>
void launchResidualRmsNormKernel(kernels::AllReduceParams& params, hipStream_t stream, AllReduceFusionOp fusionOp)
{
    if (params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, true, true, true>(params, stream, fusionOp);
    }
    else if (params.fusion_params.bias_buffer && !params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, true, true, false>(params, stream, fusionOp);
    }
    else if (!params.fusion_params.bias_buffer && params.fusion_params.weight_buffer)
    {
        reduce_fusion::rms_norm_kernel_launcher<T, false, true, true>(params, stream, fusionOp);
    }
    else
    {
        reduce_fusion::rms_norm_kernel_launcher<T, false, true, false>(params, stream, fusionOp);
    }
}

void residualRmsNorm(
    kernels::AllReduceParams& params, nvinfer1::DataType dataType, hipStream_t stream, AllReduceFusionOp fusionOp)
{
    sync_check_cuda_error(stream);
    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: launchResidualRmsNormKernel<float>(params, stream, fusionOp); break;
    case nvinfer1::DataType::kHALF: launchResidualRmsNormKernel<half>(params, stream, fusionOp); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: launchResidualRmsNormKernel<__hip_bfloat16>(params, stream, fusionOp); break;
#endif
    default: TLLM_THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error(stream);
}

void lamportInitialize(void* buffer, size_t size, nvinfer1::DataType dataType, hipStream_t stream)
{
    sync_check_cuda_error(stream);
    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT:
        reduce_fusion::lamport_initialize_kernel_launcher<float>(buffer, size, stream);
        break;
    case nvinfer1::DataType::kHALF:
        reduce_fusion::lamport_initialize_kernel_launcher<half>(buffer, size, stream);
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        reduce_fusion::lamport_initialize_kernel_launcher<__hip_bfloat16>(buffer, size, stream);
        break;
#endif
    default: TLLM_THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error(stream);
}

} // namespace tensorrt_llm::kernels
