#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <algorithm>
#include <cmath>
#include <cstdint>
#include <map>
#include <numeric>

#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/groupRmsNormKernels/groupRmsNormKernels.h"

namespace tensorrt_llm::kernels::group_rms_norm
{
// Helper function to calculate the number of warps to launch for GroupRMSNormBase
template <typename DType, int n>
uint32_t calculateNumWarpsBase(GroupRMSParams<n> const& params)
{
    constexpr uint32_t kPackedSize = sizeof(float4) / sizeof(DType);
    uint32_t input_chunk_per_warp = 32 * kPackedSize;

    // Calculate rounded input dimensions and total input length
    int rounded_input_dims[n];
    for (uint32_t i = 0; i < params.num_inputs; i++)
    {
        // Make rounded_input_dims[i] a multiple of 32 * kPackedSize
        rounded_input_dims[i]
            = (params.input_last_dims[i] + input_chunk_per_warp - 1) / input_chunk_per_warp * input_chunk_per_warp;
    }

    // Calculate total warps needed
    uint32_t total_input_length = std::accumulate(rounded_input_dims, rounded_input_dims + params.num_inputs, 0);
    uint32_t total_warps_needed = total_input_length / input_chunk_per_warp;
    return std::min<uint32_t>(32, total_warps_needed);
}

template <typename DType, int n>
struct LargeBatchWarpsInfo
{
    uint32_t num_warps_to_launch;   // Total warps to launch
    uint32_t num_warps_to_launch_0; // Warps for first input
    uint32_t num_warps_to_launch_1; // Warps for second input
    uint32_t rounds_0;              // Rounds for first input
    uint32_t rounds_1;              // Rounds for second input
};

// Helper function to calculate the number of warps to launch for GroupRMSNormKernelLargeBatch
template <typename DType, int n>
LargeBatchWarpsInfo<DType, n> calculateNumWarpsLargeBatch(GroupRMSParams<n> const& params)
{
    constexpr uint32_t kPackedSize = sizeof(float4) / sizeof(DType);
    uint32_t input_chunk_per_warp = 32 * kPackedSize;

    // Calculate warps needed for each input
    uint32_t warps_needed_0 = (params.input_last_dims[0] + input_chunk_per_warp - 1) / input_chunk_per_warp;
    uint32_t warps_needed_1 = (params.input_last_dims[1] + input_chunk_per_warp - 1) / input_chunk_per_warp;

    LargeBatchWarpsInfo<DType, n> info;
    info.num_warps_to_launch_0 = std::min((uint32_t) 32, warps_needed_0);
    info.num_warps_to_launch_1 = std::min((uint32_t) 32, warps_needed_1);

    // Use the maximum of the two for the final warps to launch
    info.num_warps_to_launch = std::max(info.num_warps_to_launch_0, info.num_warps_to_launch_1);

    // Calculate rounds needed for each input
    info.rounds_0 = (warps_needed_0 + info.num_warps_to_launch_0 - 1) / info.num_warps_to_launch_0;
    info.rounds_1 = (warps_needed_1 + info.num_warps_to_launch_1 - 1) / info.num_warps_to_launch_1;

    return info;
}

// Allocate more warps to deal with the second input
template <typename DType, typename PackedType, int n, bool EnableWeights, bool MultiRounds>
__global__ void GroupRMSNormBaseKernel(GroupRMSParams<n> params, int rounds)
{
    const uint32_t batch_idx = blockIdx.x; // Maps to batch size
    constexpr uint32_t warp_size = 32;
    const uint32_t warp_idx = threadIdx.y;
    const uint32_t lane_idx = threadIdx.x;

    static constexpr int kPackedSize = sizeof(PackedType) / sizeof(DType);

    // Each thread calculates its own partial sum
    alignas(128) __shared__ float smem_rsqrts[32];
    alignas(128) __shared__ uint32_t smem_input_mask[32];
    alignas(128) __shared__ float smem_warp_sum_sqs[32];
    if (warp_idx == 0)
    {
        smem_rsqrts[lane_idx] = 0.0f;
        // Initialize input mask to 33 to indicate no input
        smem_input_mask[lane_idx] = 33;
        smem_warp_sum_sqs[lane_idx] = 0.0f;
    }
    float warp_acc = 0.0f;
    PackedType const* __restrict__ weight_ptr = nullptr;

#if (__CUDACC_VER_MAJOR__ >= 12 && defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

    // Find which input current warp operates on
    const uint32_t input_idx = params.warp_input_idx[warp_idx]; // Maps to input index
    const uint32_t warp_start = params.warp_prefix_sum[input_idx];
    const uint32_t local_warp_idx = warp_idx - warp_start;

    PackedType const* __restrict__ input_ptr = params.inputs[input_idx];
    PackedType* __restrict__ output_ptr = params.outputs[input_idx];

    if constexpr (EnableWeights)
    {
        weight_ptr = params.weights[input_idx];
    }

    uint32_t block_offset = batch_idx * params.input_strides[input_idx];
    uint32_t round_offset = warp_size * kPackedSize;
    const uint32_t input_dim = params.input_last_dims[input_idx];

    uint32_t idx_round0 = block_offset + local_warp_idx * round_offset * rounds + lane_idx * kPackedSize;

    // Store the first round of data as local variable to reduce memory access
    PackedType input_cache;
    PackedType weight_cache;
    if (idx_round0 < input_dim + block_offset)
    {
        input_cache = input_ptr[idx_round0 / kPackedSize];
        if constexpr (EnableWeights)
        {
            weight_cache = weight_ptr[(idx_round0 - block_offset) / kPackedSize];
        }

#pragma unroll
        for (uint32_t j = 0; j < kPackedSize; j++)
        {
            float v = static_cast<float>(reinterpret_cast<DType*>(&input_cache)[j]);
            warp_acc += v * v;
        }
    }

    // Process round1+
    // If input dtype is fp16, round1+ is needed when input_dim > 8192, which is uncommon
    if constexpr (MultiRounds)
    {
        for (uint32_t i = 1; i < rounds; i++)
        {
            uint32_t idx
                = block_offset + local_warp_idx * round_offset * rounds + i * round_offset + lane_idx * kPackedSize;
            if (idx < input_dim + block_offset)
            {
                PackedType packed_data = input_ptr[idx / kPackedSize];
#pragma unroll
                for (uint32_t j = 0; j < kPackedSize; j++)
                {
                    float v = static_cast<float>(reinterpret_cast<DType*>(&packed_data)[j]);
                    warp_acc += v * v;
                }
            }
        }
    }
    float warp_sum = tensorrt_llm::common::warpReduceSum(warp_acc);
    if (lane_idx == 0)
    {
        smem_warp_sum_sqs[warp_idx] = warp_sum;
    }

// Extra _syncwarp for sm < 900, needed to avoid race condition on smem_input_mask write.
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ < 900)
    __syncwarp();
#endif
    if (lane_idx == 0)
    {
        smem_input_mask[warp_idx] = input_idx;
    }

    __syncthreads();

    warp_acc = 0.0f;
    // Cross wrap reduction on all inputs
    if (warp_idx < n)
    {
        // Each warp sums one input
        if (warp_idx == smem_input_mask[lane_idx])
        {
            warp_acc = smem_warp_sum_sqs[lane_idx];
        }
        float sum_sq = tensorrt_llm::common::warpReduceSum(warp_acc);
        if (lane_idx == 0)
        {
            smem_rsqrts[warp_idx] = rsqrtf(sum_sq / params.input_last_dims[warp_idx] + params.eps);
        }
    }

    __syncthreads();

    // Apply normalization
    if (idx_round0 < input_dim + block_offset)
    {
        PackedType packed_output;
        uint32_t output_idx = idx_round0 - block_offset + batch_idx * params.output_strides[input_idx];

#pragma unroll
        for (uint32_t j = 0; j < kPackedSize; j++)
        {
            if constexpr (EnableWeights)
            {
                reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                    static_cast<float>(reinterpret_cast<DType*>(&input_cache)[j]) * smem_rsqrts[input_idx]
                    * (static_cast<float>(reinterpret_cast<DType*>(&weight_cache)[j]) + params.weight_bias));
            }
            else
            {
                reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                    static_cast<float>(reinterpret_cast<DType*>(&input_cache)[j]) * smem_rsqrts[input_idx]);
            }
        }
        output_ptr[output_idx / kPackedSize] = packed_output;
    }

    if constexpr (MultiRounds)
    {
        for (uint32_t i = 1; i < rounds; i++)
        {
            uint32_t idx = idx_round0 + i * round_offset;
            if (idx < input_dim + block_offset)
            {
                PackedType packed_input = input_ptr[idx / kPackedSize];
                PackedType packed_output;
                uint32_t output_idx = idx - block_offset + batch_idx * params.output_strides[input_idx];

#pragma unroll
                for (uint32_t j = 0; j < kPackedSize; j++)
                {
                    if constexpr (EnableWeights)
                    {
                        reinterpret_cast<DType*>(&packed_output)[j]
                            = static_cast<float>(reinterpret_cast<DType*>(&packed_input)[j]) * smem_rsqrts[input_idx]
                            * (static_cast<float>(
                                   reinterpret_cast<DType const*>(&weight_ptr[(idx - block_offset) / kPackedSize])[j])
                                + params.weight_bias);
                    }
                    else
                    {
                        reinterpret_cast<DType*>(&packed_output)[j]
                            = static_cast<float>(reinterpret_cast<DType*>(&packed_input)[j]) * smem_rsqrts[input_idx];
                    }
                }
                output_ptr[output_idx / kPackedSize] = packed_output;
            }
        }
    }

#if (__CUDACC_VER_MAJOR__ >= 12 && defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

// This kernel is optimized for large batch sizes with 2 inputs
// Some warps process both inputs
// Fewer warps are launched allowing for more blocks to be scheduled on one SM
template <typename DType, typename PackedType, int n, bool EnableWeights, bool MultiRounds_0, bool MultiRounds_1>
__global__ void GroupRMSNormKernelLargeBatch(
    GroupRMSParams<n> params, int rounds_0, int rounds_1, int warp_size_0, int warp_size_1)
{
    const uint32_t batch_idx = blockIdx.x; // Maps to batch size
    constexpr uint32_t warp_size = 32;
    const uint32_t warp_idx = threadIdx.y;
    const uint32_t lane_idx = threadIdx.x;
    static constexpr int kPackedSize = sizeof(PackedType) / sizeof(DType);

    alignas(128) __shared__ float smem_rsqrts[n][32];

    if (warp_idx < n)
    {
        smem_rsqrts[warp_idx][lane_idx] = 0.0f;
    }

    const uint32_t round_offset = warp_size * kPackedSize;

    float sum_sq_0 = 0.0f;
    float sum_sq_1 = 0.0f;

    // Cache for prefetching
    PackedType input_0_cache;
    PackedType input_1_cache;
    PackedType weight_0_cache;
    PackedType weight_1_cache;

    PackedType const* __restrict__ weight_ptr_0 = nullptr;
    PackedType const* __restrict__ weight_ptr_1 = nullptr;

    bool process_input_0 = warp_idx < warp_size_0;
    bool process_input_1 = warp_idx < warp_size_1;

#if (__CUDACC_VER_MAJOR__ >= 12 && defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

    // Get input pointers
    PackedType const* __restrict__ input_ptr_0 = params.inputs[0];
    PackedType* __restrict__ output_ptr_0 = params.outputs[0];
    PackedType const* __restrict__ input_ptr_1 = params.inputs[1];
    PackedType* __restrict__ output_ptr_1 = params.outputs[1];

    if constexpr (EnableWeights)
    {
        weight_ptr_0 = params.weights[0];
        weight_ptr_1 = params.weights[1];
    }

    const uint32_t block_offset_0 = batch_idx * params.input_strides[0];
    const uint32_t block_offset_1 = batch_idx * params.input_strides[1];

    const uint32_t input_dim_0 = params.input_last_dims[0];
    const uint32_t input_dim_1 = params.input_last_dims[1];

    uint32_t idx_0 = block_offset_0 + warp_idx * round_offset * rounds_0 + lane_idx * kPackedSize;
    uint32_t idx_1 = block_offset_1 + warp_idx * round_offset * rounds_1 + lane_idx * kPackedSize;

    // Prefetch inputs for round0
    if (idx_0 < block_offset_0 + input_dim_0)
    {
        input_0_cache = input_ptr_0[idx_0 / kPackedSize];
        if constexpr (EnableWeights)
        {
            weight_0_cache = weight_ptr_0[(idx_0 - block_offset_0) / kPackedSize];
        }
    }
    if (idx_1 < block_offset_1 + input_dim_1)
    {
        input_1_cache = input_ptr_1[idx_1 / kPackedSize];
        if constexpr (EnableWeights)
        {
            weight_1_cache = weight_ptr_1[(idx_1 - block_offset_1) / kPackedSize];
        }
    }

    // Process round0
    if (idx_0 < block_offset_0 + input_dim_0)
    {
#pragma unroll
        for (uint32_t j = 0; j < kPackedSize; j++)
        {
            float val = static_cast<float>(reinterpret_cast<DType*>(&input_0_cache)[j]);
            sum_sq_0 += val * val;
        }
    }

    // Process round1+
    // If input dtype is fp16, round1+ is needed when input_dim > 8192, which is uncommon
    if constexpr (MultiRounds_0)
    {
        for (uint32_t i = 1; i < rounds_0; i++)
        {
            uint32_t idx
                = block_offset_0 + warp_idx * round_offset * rounds_0 + i * round_offset + lane_idx * kPackedSize;
            if (idx < block_offset_0 + input_dim_0)
            {
                PackedType packed_data = input_ptr_0[idx / kPackedSize];
#pragma unroll
                for (uint32_t j = 0; j < kPackedSize; j++)
                {
                    float val = static_cast<float>(reinterpret_cast<DType*>(&packed_data)[j]);
                    sum_sq_0 += val * val;
                }
            }
        }
    }

    if (process_input_0)
    {
        float warp_sum = tensorrt_llm::common::warpReduceSum(sum_sq_0);
        if (lane_idx == 0)
        {
            smem_rsqrts[0][warp_idx] = warp_sum;
        }
    }

    // Process round0
    if (idx_1 < block_offset_1 + input_dim_1)
    {
#pragma unroll
        for (uint32_t j = 0; j < kPackedSize; j++)
        {
            float val = static_cast<float>(reinterpret_cast<DType*>(&input_1_cache)[j]);
            sum_sq_1 += val * val;
        }
    }

    // Process round1+
    if constexpr (MultiRounds_1)
    {
        for (uint32_t i = 1; i < rounds_1; i++)
        {
            uint32_t idx
                = block_offset_1 + warp_idx * round_offset * rounds_1 + i * round_offset + lane_idx * kPackedSize;
            if (idx < block_offset_1 + input_dim_1)
            {
                PackedType packed_data = input_ptr_1[idx / kPackedSize];
#pragma unroll
                for (uint32_t j = 0; j < kPackedSize; j++)
                {
                    float val = static_cast<float>(reinterpret_cast<DType*>(&packed_data)[j]);
                    sum_sq_1 += val * val;
                }
            }
        }
    }

    // Store warp reduction to shared memory
    if (process_input_1)
    {
        float warp_sum = tensorrt_llm::common::warpReduceSum(sum_sq_1);
        if (lane_idx == 0)
        {
            smem_rsqrts[1][warp_idx] = warp_sum;
        }
    }

    __syncthreads();

    // The if-elseif code block is faster than if (warp_idx < 2)
    if (warp_idx == 0)
    {
        // Final reduction across warps
        float block_sum = tensorrt_llm::common::warpReduceSum(smem_rsqrts[0][lane_idx]);

        // Compute rsqrt
        if (lane_idx == 0)
        {
            smem_rsqrts[0][0] = rsqrtf(block_sum / input_dim_0 + params.eps);
        }
    }
    else if (warp_idx == 1)
    {
        float block_sum = tensorrt_llm::common::warpReduceSum(smem_rsqrts[1][lane_idx]);
        // Compute rsqrt
        if (lane_idx == 0)
        {
            smem_rsqrts[1][0] = rsqrtf(block_sum / input_dim_1 + params.eps);
        }
    }

    __syncthreads();

    // Apply normalization
    if (idx_0 < block_offset_0 + input_dim_0)
    {
        PackedType packed_output;
        uint32_t output_idx_0 = idx_0 - block_offset_0 + batch_idx * params.output_strides[0];

#pragma unroll
        for (uint32_t j = 0; j < kPackedSize; j++)
        {
            if constexpr (EnableWeights)
            {
                reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                    static_cast<float>(reinterpret_cast<DType*>(&input_0_cache)[j]) * smem_rsqrts[0][0]
                    * (static_cast<float>(reinterpret_cast<DType*>(&weight_0_cache)[j]) + params.weight_bias));
            }
            else
            {
                reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                    static_cast<float>(reinterpret_cast<DType*>(&input_0_cache)[j]) * smem_rsqrts[0][0]);
            }
        }
        output_ptr_0[output_idx_0 / kPackedSize] = packed_output;
    }

    if constexpr (MultiRounds_0)
    {
        for (uint32_t i = 1; i < rounds_0; i++)
        {
            uint32_t idx = idx_0 + i * round_offset;
            if (idx < block_offset_0 + input_dim_0)
            {
                PackedType packed_input = input_ptr_0[idx / kPackedSize];
                PackedType packed_output;
                uint32_t output_idx_0 = idx - block_offset_0 + batch_idx * params.output_strides[0];

#pragma unroll
                for (uint32_t j = 0; j < kPackedSize; j++)
                {
                    if constexpr (EnableWeights)
                    {
                        reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                            static_cast<float>(reinterpret_cast<DType*>(&packed_input)[j]) * smem_rsqrts[0][0]
                            * (static_cast<float>(reinterpret_cast<DType const*>(
                                   &weight_ptr_0[(idx - block_offset_0) / kPackedSize])[j])
                                + params.weight_bias));
                    }
                    else
                    {
                        reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                            static_cast<float>(reinterpret_cast<DType*>(&packed_input)[j]) * smem_rsqrts[0][0]);
                    }
                }
                output_ptr_0[output_idx_0 / kPackedSize] = packed_output;
            }
        }
    }

    if (idx_1 < block_offset_1 + input_dim_1)
    {
        PackedType packed_output;
        uint32_t output_idx_1 = idx_1 - block_offset_1 + batch_idx * params.output_strides[1];
#pragma unroll
        for (uint32_t j = 0; j < kPackedSize; j++)
        {
            if constexpr (EnableWeights)
            {
                reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                    static_cast<float>(reinterpret_cast<DType*>(&input_1_cache)[j]) * smem_rsqrts[1][0]
                    * (static_cast<float>(reinterpret_cast<DType*>(&weight_1_cache)[j]) + params.weight_bias));
            }
            else
            {
                reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                    static_cast<float>(reinterpret_cast<DType*>(&input_1_cache)[j]) * smem_rsqrts[1][0]);
            }
        }
        output_ptr_1[output_idx_1 / kPackedSize] = packed_output;
    }

    if constexpr (MultiRounds_1)
    {
        for (uint32_t i = 1; i < rounds_1; i++)
        {
            uint32_t idx = idx_1 + i * round_offset;
            if (idx < block_offset_1 + input_dim_1)
            {
                PackedType packed_input = input_ptr_1[idx / kPackedSize];
                PackedType packed_output;
                uint32_t output_idx_1 = idx - block_offset_1 + batch_idx * params.output_strides[1];

#pragma unroll
                for (uint32_t j = 0; j < kPackedSize; j++)
                {
                    if constexpr (EnableWeights)
                    {
                        reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                            static_cast<float>(reinterpret_cast<DType*>(&packed_input)[j]) * smem_rsqrts[1][0]
                            * (static_cast<float>(reinterpret_cast<DType const*>(
                                   &weight_ptr_1[(idx - block_offset_1) / kPackedSize])[j])
                                + params.weight_bias));
                    }
                    else
                    {
                        reinterpret_cast<DType*>(&packed_output)[j] = static_cast<DType>(
                            static_cast<float>(reinterpret_cast<DType*>(&packed_input)[j]) * smem_rsqrts[1][0]);
                    }
                }
                output_ptr_1[output_idx_1 / kPackedSize] = packed_output;
            }
        }
    }

#if (__CUDACC_VER_MAJOR__ >= 12 && defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <typename DType, int n, bool EnableWeights>
void GroupRMSNormBaseKernel(GroupRMSParams<n>& params)
{
    // Kernel assertions
    constexpr uint32_t kPackedSize = sizeof(float4) / sizeof(DType);
    TLLM_CHECK_WITH_INFO(params.num_inputs <= 2,
        "Only up to 2 inputs are supported with performance guarantees. Kernels with more than 2 inputs can be "
        "instantiated.");
    int rounded_input_dims[n];
    uint32_t input_chunk_per_warp = 32 * kPackedSize;
    for (uint32_t i = 0; i < params.num_inputs; i++)
    {
        TLLM_CHECK_WITH_INFO(
            params.input_last_dims[i] % 32 == 0, "The last dimension of input must be divisible by 32.");
        TLLM_CHECK_WITH_INFO(params.input_last_dims[i] % kPackedSize == 0,
            "Input[%u] dimension %u is not divisible by %u (128b / sizeof(dype)). Finer granularity is not "
            "supported yet.",
            i, params.input_last_dims[i], kPackedSize);
        // Make rounded_input_dims[i] a multiple of 32 * kPackedSize
        rounded_input_dims[i]
            = (params.input_last_dims[i] + input_chunk_per_warp - 1) / input_chunk_per_warp * input_chunk_per_warp;
    }

    // Calculate total warps to launch and rounds needed
    uint32_t total_input_length = std::accumulate(rounded_input_dims, rounded_input_dims + params.num_inputs, 0);
    uint32_t total_warps_needed = total_input_length / input_chunk_per_warp;
    uint32_t num_warps_to_launch = calculateNumWarpsBase<DType, n>(params);
    uint32_t rounds = (total_warps_needed + num_warps_to_launch - 1) / num_warps_to_launch; // ceil_div

    // Calculate warp_prefix_sum
    float warps_per_token = float(num_warps_to_launch) / total_input_length;
    std::vector<int> warps_per_array(params.num_inputs);
    int warp_prefix = 0;

    for (int i = 0; i < params.num_inputs; ++i)
    {
        params.warp_prefix_sum[i] = warp_prefix;
        warps_per_array[i] = std::max(1, int(round(rounded_input_dims[i] * warps_per_token)));
        for (int j = warp_prefix; j < warp_prefix + warps_per_array[i]; ++j)
        {
            params.warp_input_idx[j] = i;
        }
        warp_prefix += warps_per_array[i];
    }
    params.warp_prefix_sum[params.num_inputs] = warp_prefix;

    dim3 grid_dim(params.batch_size);
    dim3 block_dim(32, num_warps_to_launch);

    hipLaunchConfig_t cfg;
    hipLaunchAttribute attribute[1];
    cfg.gridDim = grid_dim;
    cfg.blockDim = block_dim;
    cfg.dynamicSmemBytes = 0;
    cfg.stream = params.stream;
    attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attribute[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL() ? 1 : 0;
    cfg.attrs = attribute;
    cfg.numAttrs = 1;
    if (rounds > 1)
    {
        TLLM_CUDA_CHECK(
            cudaLaunchKernelEx(&cfg, GroupRMSNormBaseKernel<DType, float4, n, EnableWeights, true>, params, rounds));
    }
    else
    {
        TLLM_CUDA_CHECK(
            cudaLaunchKernelEx(&cfg, GroupRMSNormBaseKernel<DType, float4, n, EnableWeights, false>, params, rounds));
    }
}

template <int n>
void GroupRMSNormBaseKernelLauncher(GroupRMSParams<n>& params)
{
#define GROUP_RMS_NORM_DISPATCH(DTYPE)                                                                                 \
    if (params.enable_weights)                                                                                         \
    {                                                                                                                  \
        return GroupRMSNormBaseKernel<DTYPE, n, true>(params);                                                         \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        return GroupRMSNormBaseKernel<DTYPE, n, false>(params);                                                        \
    }

    switch (params.dtype)
    {
    case nvinfer1::DataType::kHALF: GROUP_RMS_NORM_DISPATCH(half); break;
    case nvinfer1::DataType::kBF16: GROUP_RMS_NORM_DISPATCH(__hip_bfloat16); break;
    case nvinfer1::DataType::kFLOAT: GROUP_RMS_NORM_DISPATCH(float); break;
    default: TLLM_CHECK_WITH_INFO(false, "Unsupported data type for GroupRMSNorm");
    }

#undef GROUP_RMS_NORM_DISPATCH
}

#define INSTANTIATE_GROUP_RMS_NORM_BASE(n) template void GroupRMSNormBaseKernelLauncher<n>(GroupRMSParams<n> & params);

INSTANTIATE_GROUP_RMS_NORM_BASE(1)
INSTANTIATE_GROUP_RMS_NORM_BASE(2)

template <typename DType, int n, bool EnableWeights>
void GroupRMSNormKernelLargeBatch(GroupRMSParams<n>& params)
{
    // Kernel assertions
    constexpr uint32_t kPackedSize = sizeof(float4) / sizeof(DType);
    TLLM_CHECK_WITH_INFO(params.num_inputs == 2, "GroupRMSNormKernelLargeBatch only supports exactly 2 inputs.");

    for (uint32_t i = 0; i < params.num_inputs; i++)
    {
        TLLM_CHECK_WITH_INFO(
            params.input_last_dims[i] % 32 == 0, "The last dimension of input must be divisible by 32.");
        TLLM_CHECK_WITH_INFO(params.input_last_dims[i] % kPackedSize == 0,
            "Input[%u] dimension %u is not divisible by %u (128b / sizeof(dype)). Finer granularity is not "
            "supported yet.",
            i, params.input_last_dims[i], kPackedSize);
    }

    // Calculate warps information
    auto warpInfo = calculateNumWarpsLargeBatch<DType, n>(params);
    uint32_t num_warps_to_launch = warpInfo.num_warps_to_launch;
    uint32_t rounds_0 = warpInfo.rounds_0;
    uint32_t rounds_1 = warpInfo.rounds_1;

    dim3 grid_dim(params.batch_size);
    dim3 block_dim(32, num_warps_to_launch);

    hipLaunchConfig_t cfg;
    hipLaunchAttribute attribute[1];
    cfg.gridDim = grid_dim;
    cfg.blockDim = block_dim;
    cfg.dynamicSmemBytes = 0;
    cfg.stream = params.stream;
    attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attribute[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL() ? 1 : 0;
    cfg.attrs = attribute;
    cfg.numAttrs = 1;
    bool MultiRounds_0 = rounds_0 > 1;
    bool MultiRounds_1 = rounds_1 > 1;

    // Choose kernel based on whether weights are enabled and rounds needed
    if (MultiRounds_0 && MultiRounds_1)
    {
        TLLM_CUDA_CHECK(
            cudaLaunchKernelEx(&cfg, GroupRMSNormKernelLargeBatch<DType, float4, n, EnableWeights, true, true>, params,
                rounds_0, rounds_1, warpInfo.num_warps_to_launch_0, warpInfo.num_warps_to_launch_1));
    }
    else if (MultiRounds_0 && !MultiRounds_1)
    {
        TLLM_CUDA_CHECK(
            cudaLaunchKernelEx(&cfg, GroupRMSNormKernelLargeBatch<DType, float4, n, EnableWeights, true, false>, params,
                rounds_0, rounds_1, warpInfo.num_warps_to_launch_0, warpInfo.num_warps_to_launch_1));
    }
    else if (!MultiRounds_0 && MultiRounds_1)
    {
        TLLM_CUDA_CHECK(
            cudaLaunchKernelEx(&cfg, GroupRMSNormKernelLargeBatch<DType, float4, n, EnableWeights, false, true>, params,
                rounds_0, rounds_1, warpInfo.num_warps_to_launch_0, warpInfo.num_warps_to_launch_1));
    }
    else
    {
        TLLM_CUDA_CHECK(
            cudaLaunchKernelEx(&cfg, GroupRMSNormKernelLargeBatch<DType, float4, n, EnableWeights, false, false>,
                params, rounds_0, rounds_1, warpInfo.num_warps_to_launch_0, warpInfo.num_warps_to_launch_1));
    }
}

template <int n>
void GroupRMSNormKernelLargeBatchLauncher(GroupRMSParams<n>& params)
{
#define GROUP_RMS_NORM_LARGE_BATCH_DISPATCH(DTYPE)                                                                     \
    if (params.enable_weights)                                                                                         \
    {                                                                                                                  \
        return GroupRMSNormKernelLargeBatch<DTYPE, n, true>(params);                                                   \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        return GroupRMSNormKernelLargeBatch<DTYPE, n, false>(params);                                                  \
    }

    switch (params.dtype)
    {
    case nvinfer1::DataType::kHALF: GROUP_RMS_NORM_LARGE_BATCH_DISPATCH(half); break;
    case nvinfer1::DataType::kBF16: GROUP_RMS_NORM_LARGE_BATCH_DISPATCH(__hip_bfloat16); break;
    case nvinfer1::DataType::kFLOAT: GROUP_RMS_NORM_LARGE_BATCH_DISPATCH(float); break;
    default: TLLM_CHECK_WITH_INFO(false, "Unsupported data type for GroupRMSNormV2");
    }

#undef GROUP_RMS_NORM_LARGE_BATCH_DISPATCH
}

#define INSTANTIATE_GROUP_RMS_NORM_LARGE_BATCH(n)                                                                      \
    template void GroupRMSNormKernelLargeBatchLauncher<n>(GroupRMSParams<n> & params);

INSTANTIATE_GROUP_RMS_NORM_LARGE_BATCH(2)

int getComputeCapabilityMajor()
{
    int device;
    TLLM_CUDA_CHECK(hipGetDevice(&device));
    hipDeviceProp_t prop;
    TLLM_CUDA_CHECK(hipGetDeviceProperties(&prop, device));
    return prop.major;
}

bool prefer_base_kernel(int batch, int base_warps, float scheduling_efficiency_ratio)
{
    static std::map<std::tuple<int, int, float>, bool> cache;
    auto key = std::make_tuple(batch, base_warps, scheduling_efficiency_ratio);
    if (cache.find(key) != cache.end())
    {
        return cache[key];
    }

    int sm_major = getComputeCapabilityMajor();
    bool found_match = false;
    for (auto const& [known_model, model] : gpu_models)
    {
        if (sm_major == known_model)
        {
            float p = model.batch_size * batch + model.base_warps * base_warps
                + model.scheduling_efficiency_ratio * scheduling_efficiency_ratio + model.intercept;
            p = 1.0f / (1.0f + std::exp(-p));
            found_match = true;
            cache[key] = p > 0.5f;
            return cache[key];
        }
    }
    if (!found_match)
    {
        TLLM_LOG_INFO(
            "GroupRMSNorm: Failed to find heuristic for GPU compute capability %d. Falling back to the base kernel.",
            sm_major);
    }
    cache[key] = true;
    return true;
}

template <int n>
void GroupRMSNormKernelLauncherWithHeuristic(GroupRMSParams<n>& params)
{
    if (params.num_inputs == 1)
    {
        GroupRMSNormBaseKernelLauncher<n>(params);
    }
    else if (params.num_inputs == 2)
    {
        int num_warps_per_sm = 64;
        uint32_t base_warps;
        uint32_t large_batch_warps;

        // Choose the appropriate DType
        switch (params.dtype)
        {
        case nvinfer1::DataType::kHALF:
            base_warps = calculateNumWarpsBase<half, n>(params);
            large_batch_warps = calculateNumWarpsLargeBatch<half, n>(params).num_warps_to_launch;
            break;
        case nvinfer1::DataType::kBF16:
            base_warps = calculateNumWarpsBase<__hip_bfloat16, n>(params);
            large_batch_warps = calculateNumWarpsLargeBatch<__hip_bfloat16, n>(params).num_warps_to_launch;
            break;
        case nvinfer1::DataType::kFLOAT:
            base_warps = calculateNumWarpsBase<float, n>(params);
            large_batch_warps = calculateNumWarpsLargeBatch<float, n>(params).num_warps_to_launch;
            break;
        default: TLLM_CHECK_WITH_INFO(false, "Unsupported data type for GroupRMSNorm"); return;
        }

        int concurrent_block_per_sm_base = std::floor(num_warps_per_sm / base_warps);
        int concurrent_block_per_sm_large_batch = std::floor(num_warps_per_sm / large_batch_warps);

        /*
         * Kernel Selection Logic:
         * We use trained Logistic Regression models to determine which kernel variant to use based on performance
         * characteristics:
         *
         * - base_warps: Proportional to the sum of last dimensions of inputs
         * - large_batch_warps: Proportional to the max of last dimensions of inputs
         *
         * Trade-offs:
         * - With equal concurrent blocks per SM, base_warps achieves better compute efficiency
         * - However, large_batch_warps allows more concurrent blocks to be scheduled:
         *   - concurrent_block_per_sm_base: Maximum blocks of base kernel schedulable per SM
         *   - concurrent_block_per_sm_large_batch: Maximum blocks of large batch kernel schedulable per SM
         *
         * The large batch kernel is preferred when the scheduling efficiency advantage outweighs
         * the compute efficiency advantage of the base kernel, particularly at larger batch sizes.
         */
        if (concurrent_block_per_sm_large_batch > concurrent_block_per_sm_base)
        {
            float scheduling_efficiency_ratio
                = float(concurrent_block_per_sm_large_batch) / float(concurrent_block_per_sm_base);
            if (prefer_base_kernel(params.batch_size, base_warps, scheduling_efficiency_ratio))
            {
                GroupRMSNormBaseKernelLauncher<n>(params);
            }
            else
            {
                GroupRMSNormKernelLargeBatchLauncher<n>(params);
            }
        }
        else
        {
            GroupRMSNormBaseKernelLauncher<n>(params);
        }
    }
    else
    {
        // Unsupported number of inputs
        TLLM_CHECK_WITH_INFO(false, "Unsupported number of inputs for GroupRMSNorm");
    }
}

#define INSTANTIATE_GROUP_RMS_NORM_WITH_HEURISTIC(n)                                                                   \
    template void GroupRMSNormKernelLauncherWithHeuristic<n>(GroupRMSParams<n> & params);

INSTANTIATE_GROUP_RMS_NORM_WITH_HEURISTIC(1)
INSTANTIATE_GROUP_RMS_NORM_WITH_HEURISTIC(2)

} // namespace tensorrt_llm::kernels::group_rms_norm
