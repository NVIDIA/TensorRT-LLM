#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "moePrepareKernels.h"

#include <stdio.h>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>

namespace cg = cooperative_groups;

namespace tensorrt_llm::kernels
{

namespace moe_prepare
{

__device__ __forceinline__ void st_release_sys_global(uint64_t volatile* ptr, uint64_t val)
{
    asm volatile("st.release.sys.global.u64 [%0], %1;" ::"l"(ptr), "l"(val) : "memory");
}

__device__ __forceinline__ uint64_t ld_acquire_sys_global(uint64_t volatile* ptr)
{
    uint64_t ret;
    asm volatile("ld.acquire.sys.global.u64 %0, [%1];" : "=l"(ret) : "l"(ptr));
    return ret;
}

__device__ __forceinline__ int ld_acquire_sys_global_int(int volatile* ptr)
{
    int ret;
    asm volatile("ld.acquire.sys.global.s32 %0, [%1];" : "=r"(ret) : "l"(ptr));
    return ret;
}

class StepCommunicatorBase
{
public:
    static constexpr int META_SIZE = sizeof(MoeCommFifoConnInfo);

    __device__ __inline__ StepCommunicatorBase(MoeCommFifoConnInfo* fifoConnInfo)
        : fifoConnInfo(fifoConnInfo)
        , localCachedHead(0)
        , localCachedTail(0)
    {
    }

    __forceinline__ __device__ void reset()
    {
        fifoConnInfo->head = 0;
        fifoConnInfo->tail = 0;
    }

    __forceinline__ __device__ void releaseSendStep()
    {
        localCachedHead += 1;
        st_release_sys_global(&(fifoConnInfo->head), uint64_t(localCachedHead));
    }

    __forceinline__ __device__ void releaseRecvStep()
    {
        localCachedTail += 1;
        st_release_sys_global(&(fifoConnInfo->tail), uint64_t(localCachedTail));
    }

    __forceinline__ __device__ uint64_t acquireTail()
    {
        uint64_t tail = ld_acquire_sys_global(&(fifoConnInfo->tail));
        localCachedTail = tail;
        return tail;
    }

    __forceinline__ __device__ uint64_t acquireHead()
    {
        uint64_t head = ld_acquire_sys_global(&(fifoConnInfo->head));
        localCachedHead = head;
        return head;
    }

    __forceinline__ __device__ int acquireNewSendStep()
    {

        int64_t tail;
        do
        {
            tail = acquireTail();
        } while (localCachedHead >= tail + STEP_DEPTH);
        // depth = 2, head = 1, tail = 0 , ok
        // depth = 2, head = 2, tail = 0, should wait

        return localCachedHead % STEP_DEPTH;
    }

    __forceinline__ __device__ int acquireNewRecvStep()
    {
        int64_t head = 0;
        do
        {
            head = acquireHead();
        } while (localCachedTail >= head);

        return localCachedTail % STEP_DEPTH;
    }

public:
    MoeCommFifoConnInfo* fifoConnInfo;
    uint64_t localCachedHead;
    uint64_t localCachedTail;
    int rank;
    int targetRank;
};

// Use MoeCommFifoConnInfo as media to transfer a counter number.
// Use the "head" field as flag.
// Use the "tail" field to transfer the counter number.
class CounterCommunicator
{
public:
    __device__ __inline__ CounterCommunicator(MoeCommFifoConnInfo* fifoConnInfo)
        : fifoConnInfo(fifoConnInfo)
    {
    }

    __forceinline__ __device__ void releaseValue(uint64_t value)
    {
        // Avoid block on 0
        st_release_sys_global(&(fifoConnInfo->count), value + 1);
    }

    __forceinline__ __device__ uint64_t acquireValue()
    {
        uint64_t localCount = 0;
        do
        {
            localCount = ld_acquire_sys_global(&(fifoConnInfo->count));
        } while (localCount == 0);

        fifoConnInfo->count = 0; // reset the count

        return localCount - 1;
    }

protected:
    MoeCommFifoConnInfo* fifoConnInfo;
};

template <int kThreadsGroupSize>
__device__ __forceinline__ void computeCountAndSend(int* experts, int tokenCount, int* sharedSendRecvRankCount,
    int* sendCounts, int* sendIndiceWorkspace, int* backwardIndiceWorkspace, MoeCommWorkspace workspace,
    int maxTokenCountPerRank, int expertCount, int topK, int epRank, int epSize)
{
    cg::thread_block_tile<kThreadsGroupSize> tile = cg::tiled_partition<kThreadsGroupSize>(cg::this_thread_block());
    int laneInTile = tile.thread_rank();
    int tileId = threadIdx.x / kThreadsGroupSize;
    int tileCountPerBlock = blockDim.x / kThreadsGroupSize;
    int expertCountPerRank = expertCount / epSize;
    if (threadIdx.x == 0)
    {
        *sharedSendRecvRankCount = 0;
    }
    __syncthreads();
    int targetRankId = blockIdx.x;
    int readRankTokenCount = tokenCount;
    if (targetRankId >= epSize)
    {
        return;
    }

    int* localSendIndice = sendIndiceWorkspace + targetRankId * maxTokenCountPerRank;
    int* localBackwardIndice = backwardIndiceWorkspace + targetRankId * maxTokenCountPerRank;

    for (int i = tileId; i < readRankTokenCount; i += tileCountPerBlock)
    {
        int expertRankId = laneInTile < topK ? experts[i * topK + laneInTile] / expertCountPerRank : epSize;
        bool rankMatched = (expertRankId == targetRankId);
        bool hasRankMatched = tile.any(rankMatched);
        int mask = tile.ballot(rankMatched);
        int firstMatchLane = __ffs(mask) - 1; // only valid if hasRankMatched is true
        if (hasRankMatched && laneInTile == 0)
        {
            int index = atomicAdd_block(sharedSendRecvRankCount, 1);
            localSendIndice[index] = i;
            localBackwardIndice[index] = i * topK + firstMatchLane;
        }
        tile.sync();
    }
    __syncthreads();
    if (threadIdx.x == 0)
    {
        CounterCommunicator counter(workspace.getFifoConnInfo(true, epRank, targetRankId, 0, epSize, 1));
        int count = *(sharedSendRecvRankCount);
        // printf("sendRecvCount: %d, rankId: %d, targetRankId: %d\n", count, rankId, targetRankId);
        counter.releaseValue(uint64_t(count));
        *(sendCounts + targetRankId) = count;
    }
}

__device__ __forceinline__ void recvCount(int* recvIndiceWorkspace, int* recvCounts, int* sharedCountsBase,
    MoeCommWorkspace workspace, int maxTokenCountPerRank, int rankId, int rankCount)
{
    int rankOffset = threadIdx.x / THREADS_PER_PIPELINE;
    if (rankOffset >= PIPELINE_PER_CTA)
    {
        return;
    }
    int* sharedCountsThisRank = sharedCountsBase + rankOffset;
    int targetRankId = (blockIdx.x - rankCount) * PIPELINE_PER_CTA + rankOffset;
    if (targetRankId >= rankCount)
    {
        return;
    }
    int unitId = threadIdx.x % UNIT_PER_PIPELINE;
    cg::thread_block_tile<THREADS_PER_PIPELINE> rankTile
        = cg::tiled_partition<THREADS_PER_PIPELINE>(cg::this_thread_block());
    int* localRecvIndice = recvIndiceWorkspace + targetRankId * maxTokenCountPerRank;
    int rankRecvCount;
    if (rankTile.thread_rank() == 0)
    {
        CounterCommunicator counter(workspace.getFifoConnInfo(false, rankId, targetRankId, 0, rankCount, 1));
        rankRecvCount = int(counter.acquireValue());
        // printf("rankRecvCount: %d, rankId: %d, targetRankId: %d\n", rankRecvCount, rankId, targetRankId);
        *(recvCounts + targetRankId) = rankRecvCount;
        *(sharedCountsThisRank) = rankRecvCount;
    }
    rankTile.sync();

    rankRecvCount = *(sharedCountsThisRank);
    for (int tokenId = unitId; tokenId < rankRecvCount; tokenId += UNIT_PER_PIPELINE)
    {
        *(localRecvIndice + tokenId) = tokenId;
    }
}

template <int kThreadsGroupSize>
__global__ void computeCountAndIndiceDevice(int* experts, int* sendCounts, int* recvCounts, int* sendIndiceWorkspace,
    int* backwardIndiceWorkspace, int* recvIndiceWorkspace, MoeCommWorkspace workspace, int tokenCount,
    int maxTokenCountPerRank, int topK, int expertCount, int rankId, int rankCount)
{
    __shared__ int sharedCounts[PIPELINE_PER_CTA];
    bool isSender = blockIdx.x < rankCount;
    if (isSender)
    {
        computeCountAndSend<kThreadsGroupSize>(experts, tokenCount, &sharedCounts[0], sendCounts, sendIndiceWorkspace,
            backwardIndiceWorkspace, workspace, maxTokenCountPerRank, expertCount, topK, rankId, rankCount);
    }
    else
    {
        recvCount(
            recvIndiceWorkspace, recvCounts, &sharedCounts[0], workspace, maxTokenCountPerRank, rankId, rankCount);
    }
}

__global__ void moveIndiceDevice(int* sendCountsCumsum, int* recvCountsCumsum, int* sendIndice, int* gatherSendIndice,
    int* backwardIndice, int* gatherBackwardIndice, int* recvIndice, int* gatherRecvIndice, int maxTokenCountPerRank)
{
    int targetRankId = blockIdx.x;
    if (blockIdx.y == 0)
    {
        // sendIndice and backwardIndice CTA
        int startIndex = targetRankId == 0 ? 0 : sendCountsCumsum[targetRankId - 1];
        int endIndex = sendCountsCumsum[targetRankId];
        int count = endIndex - startIndex;
        int* localSendIndice = sendIndice + targetRankId * maxTokenCountPerRank;
        int* localBackwardIndice = backwardIndice + targetRankId * maxTokenCountPerRank;
        for (int localIdx = threadIdx.x; localIdx < count; localIdx += blockDim.x)
        {
            gatherSendIndice[startIndex + localIdx] = localSendIndice[localIdx];
            gatherBackwardIndice[startIndex + localIdx] = localBackwardIndice[localIdx];
        }
    }
    else
    {
        // recvIndice CTA
        int startIndex = targetRankId == 0 ? 0 : recvCountsCumsum[targetRankId - 1];
        int endIndex = recvCountsCumsum[targetRankId];
        int count = endIndex - startIndex;
        for (int localIdx = threadIdx.x; localIdx < count; localIdx += blockDim.x)
        {
            gatherRecvIndice[startIndex + localIdx] = startIndex + localIdx;
        }
    }
}

__global__ void computeCumsumDevice(int* sendCountsCumsum, int* recvCountsCumsum, int rankId, int rankCount)
{
    int* inputOutputPtr = blockIdx.x == 0 ? sendCountsCumsum : recvCountsCumsum;

    // Use 2 block to comuteCumsum
    typedef hipcub::BlockScan<int, CUMSUM_THREADS_PER_BLOCK> BlockScan;
    __shared__ typename BlockScan::TempStorage temp_storage;

    int tid = threadIdx.x;
    int threadData = tid < rankCount ? inputOutputPtr[tid] : 0;
    int count = threadData;
    __syncthreads();

    BlockScan(temp_storage).InclusiveSum(threadData, threadData);
    if (tid < rankCount)
    {
        inputOutputPtr[tid] = threadData;
        // printf("cumsum, send? : %d, rankId:%d, tid:%d, threadData:%d, count:%d\n", blockIdx.x == 0, rankId, tid,
        // threadData, count);
    }
}

template <typename STEP_COMMUNICATOR_TYPE>
class PacketPipeline
{
public:
    __device__ __inline__ PacketPipeline(
        void* bufferBase, STEP_COMMUNICATOR_TYPE* stepCommunicator, int* sharedNewStepPtr, bool isSender)
        : bufferBase(bufferBase)
        , stepCommunicator(stepCommunicator)
        , shared_new_step(sharedNewStepPtr)
    {
        step = 0;
        needRelease = false;
        packetId = isSender ? 0 : PACKET_PER_STEP - 1;
    }

    __device__ __forceinline__ void* getFirstSendPacket()
    {
        return bufferBase;
    }

    __device__ __inline__ void* finishSendPacket(bool acquireNewStep)
    {

        packetId++;
        if (packetId < PACKET_PER_STEP)
        {
            return acquireNewStep ? bufferBase + step * PACKET_PER_STEP * PACKET_SIZE + packetId * PACKET_SIZE
                                  : nullptr;
        }

        __syncthreads();
        if (threadIdx.x == 0)
        {
            stepCommunicator->releaseSendStep();
            if (acquireNewStep)
            {
                step = stepCommunicator->acquireNewSendStep();
                *(shared_new_step) = step;
            }
        }
        __syncthreads();

        if (acquireNewStep)
        {
            step = *(shared_new_step);
            packetId = 0;
            return bufferBase + step * PACKET_SIZE * PACKET_PER_STEP;
        }

        return nullptr;
    }

    __device__ __forceinline__ void* sendFinalize()
    {
        if (packetId > 0 && threadIdx.x == 0)
        {
            stepCommunicator->releaseSendStep();
        }
    }

    __device__ __inline__ void* getNewRecvPacket()
    {
        packetId++;
        if (packetId < PACKET_PER_STEP)
        {
            return bufferBase + step * PACKET_PER_STEP * PACKET_SIZE + packetId * PACKET_SIZE;
        }

        __syncthreads();
        if (threadIdx.x == 0)
        {
            if (needRelease)
            {
                stepCommunicator->releaseRecvStep();
            }
            step = stepCommunicator->acquireNewRecvStep();
            needRelease = true;
            *(shared_new_step) = step;
        }
        __syncthreads();
        packetId = 0;
        step = *(shared_new_step);
        void* packetPtr = bufferBase + step * PACKET_SIZE * PACKET_PER_STEP;

        return packetPtr;
    }

    __device__ __forceinline__ void reset()
    {
        if (threadIdx.x == 0)
        {
            stepCommunicator->reset();
        }
    }

    void* bufferBase;
    STEP_COMMUNICATOR_TYPE* stepCommunicator;
    int step;
    int packetId;
    bool needRelease;
    int* shared_new_step;
};

template <typename STEP_COMMUNICATOR_TYPE>
__global__ void allToAllMetadataDevice(int* sendExperts, int* recvExperts, float* sendScales, float* recvScales,
    int* localExpertStatics, int* gatheredExpertStatics, MoeCommWorkspace workspace, int* sendCountsCumsum,
    int* localSendIndice, int* recvCountsCumsum, int* localRecvIndice, int tokenCount, int maxTokenCountPerRank,
    int topK, int expertCount, int slotCount, int rankId, int rankCount)
{
    bool isSender = (blockIdx.y == 0);
    int targetRankId = blockIdx.x;
    int slotCountPerRank = slotCount / rankCount;
    int groupSize = topK / UNIT_SIZE;
    int groupId = threadIdx.x % groupSize;

    __shared__ int sharedNewStep;
    __align__(16) int experts[UNIT_SIZE];
    __align__(16) float scales[UNIT_SIZE];

    uint8_t* bufferBase = (uint8_t*) (workspace.getFifoBasePtr(isSender, rankId, targetRankId, 0, 1));
    STEP_COMMUNICATOR_TYPE stepCommunicator(workspace.getFifoConnInfo(isSender, rankId, targetRankId, 0, rankCount, 1));
    PacketPipeline<STEP_COMMUNICATOR_TYPE> pipeline(bufferBase, &stepCommunicator, &sharedNewStep, isSender);

    if (isSender)
    {
        int baseCumsum = targetRankId == 0 ? 0 : *(sendCountsCumsum + targetRankId - 1);
        int sendTokenCount = *(sendCountsCumsum + targetRankId) - baseCumsum;
        int unitCount = sendTokenCount * topK / UNIT_SIZE;

        void* packPtr = pipeline.getFirstSendPacket();
        int indexBase = 0;
        int staticCopyBase = 0;
        bool acquireNewStep = unitCount > 0 || (localExpertStatics != nullptr && expertCount > 0);
        while (acquireNewStep)
        {
            if (threadIdx.x < UNIT_PER_ITER)
            {
                int index = indexBase + threadIdx.x;
                if (index < unitCount)
                {
                    int tokenId = *(localSendIndice + maxTokenCountPerRank * targetRankId + (index / groupSize));
                    *((int4*) (experts)) = *(int4*) (sendExperts + tokenId * topK + groupId * UNIT_SIZE);
                    *((float4*) (scales)) = *(float4*) (sendScales + tokenId * topK + groupId * UNIT_SIZE);

#pragma unroll
                    for (int j = 0; j < UNIT_SIZE; j++)
                    {
                        int expertId = experts[j];
                        if (expertId / slotCountPerRank != targetRankId)
                        {
                            experts[j] = slotCount;
                            scales[j] = 0.0f;
                        }
                    }

                    int* expertsPtr = (int*) (packPtr) + threadIdx.x * UNIT_SIZE;
                    float* scaleBasePtr = (float*) (packPtr + SCALE_OFFSET);
                    float* scalesPtr = (float*) (scaleBasePtr) + threadIdx.x * UNIT_SIZE;
                    *((int4*) (expertsPtr)) = *((int4*) (experts));
                    *((float4*) (scalesPtr)) = *((float4*) (scales));
                }
            }
            else if (localExpertStatics != nullptr)
            {
                int staticCopyIdx = threadIdx.x - UNIT_PER_ITER;
                if (staticCopyBase + staticCopyIdx * 4 < expertCount)
                {
                    int4* staticBasePtr = (int4*) (packPtr + STATIC_COPY_OFFSET);
                    int4 staticData = *(int4*) (localExpertStatics + staticCopyBase + staticCopyIdx * 4);
                    *(staticBasePtr + staticCopyIdx) = staticData;
                }
            }

            indexBase += UNIT_PER_ITER;
            staticCopyBase += STATIC_COPY_PER_ITER * 4;
            acquireNewStep = indexBase < unitCount || staticCopyBase < expertCount;
            packPtr = pipeline.finishSendPacket(acquireNewStep);
        }

        pipeline.sendFinalize();
    }
    else
    {
        int baseCumsum = targetRankId == 0 ? 0 : *(recvCountsCumsum + targetRankId - 1);
        int recvTokenCount = *(recvCountsCumsum + targetRankId) - baseCumsum;
        int recvUnitCount = recvTokenCount * groupSize;

        int unitIdBase = 0;
        int staticCopyBase = 0;
        while (unitIdBase < recvUnitCount || (localExpertStatics != nullptr && staticCopyBase < expertCount))
        {
            void* packetPtr = pipeline.getNewRecvPacket();
            int packetUnitCount
                = unitIdBase + UNIT_PER_ITER < recvUnitCount ? UNIT_PER_ITER : recvUnitCount - unitIdBase;
            packetUnitCount = max(packetUnitCount, 0);
            if (threadIdx.x < UNIT_PER_ITER)
            {
                if (threadIdx.x < packetUnitCount)
                {
                    int* expertsPtr = (int*) (packetPtr) + threadIdx.x * UNIT_SIZE;
                    float* scaleBasePtr = (float*) (packetPtr + SCALE_OFFSET);
                    float* scalesPtr = scaleBasePtr + threadIdx.x * UNIT_SIZE;
                    *((int4*) (experts)) = *((int4*) (expertsPtr));
                    *((float4*) (scales)) = *((float4*) (scalesPtr));

                    int tokenId = baseCumsum + (unitIdBase + threadIdx.x) / groupSize;

                    int4* dstExpertsPtr = (int4*) (recvExperts + tokenId * topK + groupId * UNIT_SIZE);
                    float4* dstScalesPtr = (float4*) (recvScales + tokenId * topK + groupId * UNIT_SIZE);
                    *dstExpertsPtr = *((int4*) (experts));
                    *dstScalesPtr = *((float4*) (scales));
                }
            }
            else if (localExpertStatics != nullptr)
            {
                int staticCopyIdx = threadIdx.x - UNIT_PER_ITER;
                if (staticCopyBase + staticCopyIdx * 4 < expertCount)
                {
                    int4* staticBasePtr = (int4*) (packetPtr + STATIC_COPY_OFFSET);
                    int4 staticData = *(staticBasePtr + staticCopyIdx);
                    *(int4*) (gatheredExpertStatics + targetRankId * expertCount + staticCopyBase + staticCopyIdx * 4)
                        = staticData;
                }
            }

            unitIdBase += packetUnitCount;
            staticCopyBase += STATIC_COPY_PER_ITER * 4;
        }

        pipeline.reset();
    }
}

void computeCountAndIndice(int* experts, int* sendCounts, int* recvCounts, int* sendIndiceWorkspace,
    int* backwardIndiceWorkspace, int* recvIndiceWorkspace, MoeCommWorkspace workspace, int tokenCount,
    int maxTokenCountPerRank, int topK, int expert_count, int rankId, int rankCount, hipStream_t stream)
{
    // first rankCount CTAs for count and send, then rankCount / PIPELINE_PER_CTA CTAs only for receive
    int grid_x = rankCount + (rankCount + PIPELINE_PER_CTA - 1) / PIPELINE_PER_CTA;
    int block_size = 1024;
    dim3 block(block_size);
    dim3 grid(grid_x);
    TLLM_CHECK_WITH_INFO(topK >= 1 && topK <= 32, "Only 1 <= topK <= 32 is supported now.");
    auto* kernelFn = computeCountAndIndiceDevice<1>;
    if (topK > 16)
    {
        kernelFn = computeCountAndIndiceDevice<32>;
    }
    else if (topK > 8)
    {
        kernelFn = computeCountAndIndiceDevice<16>;
    }
    else if (topK > 4)
    {
        kernelFn = computeCountAndIndiceDevice<8>;
    }
    else if (topK > 2)
    {
        kernelFn = computeCountAndIndiceDevice<4>;
    }
    else if (topK > 1)
    {
        kernelFn = computeCountAndIndiceDevice<2>;
    }
    kernelFn<<<grid, block, 0, stream>>>(experts, sendCounts, recvCounts, sendIndiceWorkspace, backwardIndiceWorkspace,
        recvIndiceWorkspace, workspace, tokenCount, maxTokenCountPerRank, topK, expert_count, rankId, rankCount);
}

void computeCumsum(int* sendCountsCumsum, int* recvCountsCumsum, int rankId, int rankCount, hipStream_t stream)
{
    int block_size = CUMSUM_THREADS_PER_BLOCK;
    dim3 block(block_size);
    dim3 grid(2);
    computeCumsumDevice<<<grid, block, 0, stream>>>(sendCountsCumsum, recvCountsCumsum, rankId, rankCount);
}

void moveIndice(int* sendCountsCumsum, int* recvCountsCumsum, int* sendIndice, int* gatherSendIndice,
    int* backwardIndice, int* gatherBackwardIndice, int* recvIndice, int* gatherRecvIndice, int rankId, int rankCount,
    int maxTokenCountPerRank, hipStream_t stream)
{
    dim3 block(512);
    dim3 grid(rankCount, 2);
    moveIndiceDevice<<<grid, block, 0, stream>>>(sendCountsCumsum, recvCountsCumsum, sendIndice, gatherSendIndice,
        backwardIndice, gatherBackwardIndice, recvIndice, gatherRecvIndice, maxTokenCountPerRank);
}

void allToAllMetadata(int* sendExperts, int* recvExperts, float* sendScales, float* recvScales, int* localExpertStatics,
    int* gatheredExpertStatics, MoeCommWorkspace workspace, int* sendCountsCumsum, int* localSendIndice,
    int* recvCountsCumsum, int* localRecvIndice, int tokenCount, int maxTokenCountPerRank, int topK, int expertCount,
    int slotCount, int rankId, int rankCount, hipStream_t stream)
{
    int block_size = localExpertStatics == nullptr ? UNIT_PER_ITER : UNIT_PER_ITER + STATIC_COPY_PER_ITER;
    dim3 block(block_size);
    dim3 grid(rankCount, 2);
    assert(topK == 8);
    allToAllMetadataDevice<StepCommunicatorBase><<<grid, block, 0, stream>>>(sendExperts, recvExperts, sendScales,
        recvScales, localExpertStatics, gatheredExpertStatics, workspace, sendCountsCumsum, localSendIndice,
        recvCountsCumsum, localRecvIndice, tokenCount, maxTokenCountPerRank, topK, expertCount, slotCount, rankId,
        rankCount);
}

size_t getMoePrepareWorkspaceSize(int epSize)
{
    return (STEP_DEPTH * PACKET_PER_STEP * PACKET_SIZE + StepCommunicatorBase::META_SIZE) * epSize;
}

} // namespace moe_prepare

} // namespace tensorrt_llm::kernels
