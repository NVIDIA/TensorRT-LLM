#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/kernels/decodingCommon.h"

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/runtime/common.h"

#include <cstdint>

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::kernels
{

__global__ void curandInitialize(hiprandState_t* state, int const* batchSlots, int const size, uint64_t const randomSeed)
{
    int const idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[idx] : idx;
        hiprand_init(randomSeed, 0, 0, &state[batchSlot]);
    }
}

void invokeCurandInitialize(
    hiprandState_t* state, int const* batchSlots, size_t const batchSize, uint64_t const randomSeed, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((int) (ceil(batchSize * 1.0 / 256)));
    curandInitialize<<<grid, block, 0, stream>>>(state, batchSlots, batchSize, randomSeed);
}

__global__ void curandBatchInitialize(
    hiprandState_t* states, SizeType32 const* batchSlots, SizeType32 const size, uint64_t const* randomSeeds)
{
    SizeType32 const bid = threadIdx.x + blockIdx.x * blockDim.x;
    if (bid < size)
    {
        auto const batchSlot = batchSlots != nullptr ? batchSlots[bid] : bid;
        hiprand_init(randomSeeds[bid], 0, 0, &states[batchSlot]);
    }
}

void invokeCurandBatchInitialize(hiprandState_t* states, SizeType32 const* batchSlots, size_t const batchSize,
    uint64_t const* randomSeeds, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid(static_cast<SizeType32>(ceil(batchSize * 1.0 / 256)));
    curandBatchInitialize<<<grid, block, 0, stream>>>(states, batchSlots, batchSize, randomSeeds);
}

template <typename T>
__global__ void addBiasSoftMax(T* logits, T** logitsPtrs, T* probs, float* outputEntropy, T const* bias,
    float const* temperatures, int32_t const* endIds, FinishedState const* finished, int32_t const* beamWidths,
    int32_t const* batchSlots, float const* minPs, int32_t maxBatchSize, int32_t maxBeamWidth, int32_t vocabSize,
    int32_t vocabSizePadded, bool skipSoftMax, bool batchSlotsLogits, bool ptrsForBeams, bool const* skipDecode)
{
    auto const batchIdx = blockIdx.x;
    auto const beamIdx = blockIdx.y;
    auto const batchSlot = batchSlots ? batchSlots[batchIdx] : batchIdx;
    if (beamWidths && beamIdx >= beamWidths[batchSlot])
    {
        return;
    }
    if ((skipDecode != nullptr && skipDecode[batchSlot]))
    {
        return;
    }

    auto const batchIdxLogits = batchSlotsLogits ? batchSlot : batchIdx;
    FinishedState const finishState
        = finished != nullptr ? finished[beamIdx * maxBatchSize + batchSlot] : FinishedState::empty();
    if (finishState.isSkipDecoding())
    {
        return;
    }
    bool const finish = finishState.isFinished();

    auto logitsPtr = logitsPtrs ? (ptrsForBeams ? logitsPtrs[batchIdx * maxBeamWidth + beamIdx]
                                                : logitsPtrs[batchIdx] + beamIdx * vocabSizePadded)
                                : logits + (batchIdxLogits * maxBeamWidth + beamIdx) * vocabSizePadded;

    T const MAX_T_VAL = (std::is_same<T, half>::value) ? HALF_FLT_MAX : FLT_MAX;
    float const EPSILON = (std::is_same<T, half>::value) ? 1e-3f : 1e-6f;
    float maxVal = -FLT_MAX;
    __shared__ float sMaxVal, sSumVal;

    auto const tempInv = temperatures ? T{1.f / (temperatures[batchSlot] + EPSILON)} : T{1.f};

    for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
    {
        auto logit = logitsPtr[tid];
        logit = temperatures ? logit * tempInv : logit;
        if (tid < vocabSize)
        {
            if (finish && endIds != nullptr)
            {
                // Prefer token EOS if the request has finished
                logit = (tid == endIds[batchSlot]) ? MAX_T_VAL : -MAX_T_VAL;
            }
            else
            {
                // Compute biased logit if the request has not finished, or `endIds` is nullptr
                logit += (bias != nullptr) ? bias[tid] : T{0.0f};
            }
        }
        else
        {
            logit = -MAX_T_VAL;
        }
        maxVal = max(maxVal, static_cast<float>(logit));
        logitsPtr[tid] = logit; // Write back biased logits
    }

    float minP = minPs != nullptr ? minPs[batchSlot] : 0.0f;

    if (!skipSoftMax)
    {
        maxVal = blockReduceMax<float>(static_cast<float>(maxVal));
        if (threadIdx.x == 0)
        {
            sMaxVal = maxVal;
        }
        __syncthreads();

        // `probs == nullptr` is specialization for Beam-Search, which needs log and writes output to`logitsPtrs`
        float sumVal = 0.0f;
        int const offset = (probs != nullptr) ? ((batchIdxLogits * maxBeamWidth + beamIdx) * vocabSizePadded) : 0;
        T* dst = (probs != nullptr) ? probs : logitsPtr;
        for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
        {
            auto value = __expf(static_cast<float>(logitsPtr[tid]) - sMaxVal);
            // minP : probability of token proportional to the max token
            // compare minP against exp(logit - maxVal) / exp(maxVal - maxVal) = exp(logit - maxVal)
            if (value < minP)
            {
                value = 0.0;
                logitsPtr[tid] = -MAX_T_VAL;
            }
            dst[offset + tid] = value;
            sumVal += value;
        }

        sumVal = blockReduceSum<float>(sumVal);
        if (threadIdx.x == 0)
        {
            sSumVal = sumVal;
        }
        __syncthreads();

        float entropy{0.f};
        for (int tid = threadIdx.x; tid < vocabSizePadded; tid += blockDim.x)
        {
            auto const softmaxValue = static_cast<float>(dst[offset + tid]) / (sSumVal + EPSILON);
            auto const probValue = (probs != nullptr) ? softmaxValue : __logf(softmaxValue);
            if (outputEntropy)
            {
                entropy += probValue * __logf(probValue + EPSILON);
            }
            dst[offset + tid] = probValue;
        }

        if (outputEntropy)
        {
            entropy = blockReduceSum<float>(entropy);

            if (threadIdx.x == 0)
            {
                outputEntropy[batchSlot * maxBeamWidth + beamIdx] = -entropy;
            }
        }
    }
}

template <typename T>
void invokeAddBiasSoftMax(BiasSoftmaxParams<T> const params, hipStream_t stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    dim3 grid(params.batchSize, params.maxBeamWidth);
    auto const vocabRoundedToWarp = roundUp(params.vocabSize, 32);
    dim3 block(std::min(vocabRoundedToWarp, 1024)); // vocabSize is usually larger than 1024
    addBiasSoftMax<<<grid, block, 0, stream>>>(params.logits, params.logitsPtrs, params.probs, params.outputEntropy,
        params.bias, params.temperatures, params.endIds, params.finished, params.beamWidths, params.batchSlots,
        params.minPs, params.maxBatchSize, params.maxBeamWidth, params.vocabSize, params.vocabSizePadded,
        params.skipSoftMax, params.batchSlotsLogits, params.ptrsForBeams, params.skipDecode);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template void invokeAddBiasSoftMax(BiasSoftmaxParams<float> const params, hipStream_t stream);
template void invokeAddBiasSoftMax(BiasSoftmaxParams<half> const params, hipStream_t stream);

template <typename T>
__global__ void scatterDecodingParamsKernel(T const* src, T scalar, T* dst, int const* batchSlots, int batchSize)
{
    auto const batchIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batchIdx >= batchSize)
    {
        return;
    }
    auto const batchSlot = batchSlots[batchIdx];
    dst[batchSlot] = (src == nullptr ? scalar : src[batchIdx]);
}

template <typename T>
void invokeScatterDecodingParams(
    T const* src, T scalar, T* dst, int const* batchSlots, int batchSize, hipStream_t stream)
{
    constexpr int THREADS_PER_CTA = 256;
    dim3 grid(divUp(batchSize, THREADS_PER_CTA));
    scatterDecodingParamsKernel<<<grid, THREADS_PER_CTA, 0, stream>>>(src, scalar, dst, batchSlots, batchSize);
}

template void invokeScatterDecodingParams(
    float const* src, float scalar, float* dst, int const* batchSlots, int batchSize, hipStream_t stream);
template void invokeScatterDecodingParams(
    uint32_t const* src, uint32_t scalar, uint32_t* dst, int const* batchSlots, int batchSize, hipStream_t stream);
template void invokeScatterDecodingParams(
    int32_t const* src, int32_t scalar, int32_t* dst, int const* batchSlots, int batchSize, hipStream_t stream);

} // namespace tensorrt_llm::kernels
