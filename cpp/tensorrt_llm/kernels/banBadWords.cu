#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/banBadWords.h"

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T>
__global__ void ban_bad_words(T* logits, TokenIdType const** output_ids_ptr, SizeType32 const** parent_ids_ptr,
    SizeType32 const* batch_slots, SizeType32 beam_width, TokenIdType const* const* bad_words_ptrs,
    SizeType32 const* bad_words_lens, SizeType32 vocab_size_padded, SizeType32 const* sequence_lengths,
    SizeType32 max_seq_len)
{
    auto const id = blockIdx.x * blockDim.x + threadIdx.x;
    auto const batch_idx = blockIdx.y / beam_width;
    auto const beam_idx = blockIdx.y % beam_width;
    auto const batch_slot = batch_slots != nullptr ? batch_slots[batch_idx] : batch_idx;
    auto const batch_beam_idx = batch_slot * beam_width + beam_idx;

    auto const* base_bad_words = bad_words_ptrs[batch_slot];
    auto const bad_words_len = bad_words_lens[batch_slot];
    auto const* base_bad_words_offsets = base_bad_words + bad_words_len;

    if (id >= bad_words_len || base_bad_words_offsets[id] < 0)
    {
        return;
    }

    auto const item_end = base_bad_words_offsets[id];
    auto const item_start = (id > 0) ? base_bad_words_offsets[id - 1] : 0;
    auto const item_size = item_end - item_start;

    /* The single-token case unconditionally bans the token */
    bool should_ban = item_size == 1;
    auto const current_step{sequence_lengths[batch_beam_idx]};
    /* Multi-token case and enough previously generated tokens to look for a match
     */
    if (item_size > 1 && current_step >= item_size - 1)
    {
        should_ban = true;
        auto parent_id = static_cast<SizeType32>(beam_idx);
        bool const gather_beam = beam_width > 1;

        for (auto token_idx = item_size - 2; token_idx >= 0; token_idx--)
        {
            auto const previous_token
                = output_ids_ptr[batch_slot][parent_id * max_seq_len + current_step - (item_size - 1) + token_idx];

            if (previous_token != base_bad_words[item_start + token_idx])
            {
                should_ban = false;
                break;
            }
            if (gather_beam)
            {
                parent_id = parent_ids_ptr == nullptr
                    ? SizeType32{0}
                    : parent_ids_ptr[batch_slot][parent_id * max_seq_len + current_step - (item_size - 1) + token_idx];

                if (parent_id < 0 || parent_id >= beam_width)
                {
                    should_ban = false;
                    break;
                }
            }
        }
    }

    if (should_ban)
    {
        auto banned_token = base_bad_words[item_end - 1];
        if (0 <= banned_token && banned_token < vocab_size_padded)
        {
            logits[batch_idx * beam_width * vocab_size_padded + beam_idx * vocab_size_padded + banned_token]
                = static_cast<T>(-INFINITY);
        }
    }
}

template <typename T>
void invokeBanBadWords(T* logits, TokenIdType const** output_ids_ptr, SizeType32 const** parent_ids_ptr,
    SizeType32 const* batch_slot, SizeType32 batch_size, SizeType32 beam_width, TokenIdType const* const* bad_words,
    SizeType32 const* bad_words_lens, SizeType32 max_bad_words_len, SizeType32 vocab_size_padded,
    SizeType32 const* sequence_lengths, SizeType32 max_seq_len, hipStream_t stream)
{
    dim3 block, grid;
    constexpr SizeType32 max_blocks{256};
    block.x = min(((max_bad_words_len + 32 - 1) / 32) * 32, max_blocks);
    grid.x = (max_bad_words_len + block.x - 1) / block.x;
    grid.y = batch_size * beam_width;

    ban_bad_words<<<grid, block, 0, stream>>>(logits, output_ids_ptr, parent_ids_ptr, batch_slot, beam_width, bad_words,
        bad_words_lens, vocab_size_padded, sequence_lengths, max_seq_len);
    sync_check_cuda_error(stream);
}

template void invokeBanBadWords(half* logits, TokenIdType const** output_ids_ptr, SizeType32 const** parent_ids_ptr,
    SizeType32 const* batch_slot, SizeType32 batch_size, SizeType32 beam_width, TokenIdType const* const* bad_words,
    SizeType32 const* bad_words_lens, SizeType32 max_bad_words_len, SizeType32 vocab_size_padded,
    SizeType32 const* sequence_lengths, SizeType32 max_seq_len, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeBanBadWords(__hip_bfloat16* logits, TokenIdType const** output_ids_ptr,
    SizeType32 const** parent_ids_ptr, SizeType32 const* batch_slot, SizeType32 batch_size, SizeType32 beam_width,
    TokenIdType const* const* bad_words, SizeType32 const* bad_words_lens, SizeType32 max_bad_words_len,
    SizeType32 vocab_size_padded, SizeType32 const* sequence_lengths, SizeType32 max_seq_len, hipStream_t stream);
#endif
template void invokeBanBadWords(float* logits, TokenIdType const** output_ids_ptr, SizeType32 const** parent_ids_ptr,
    SizeType32 const* batch_slot, SizeType32 batch_size, SizeType32 beam_width, TokenIdType const* const* bad_words,
    SizeType32 const* bad_words_lens, SizeType32 max_bad_words_len, SizeType32 vocab_size_padded,
    SizeType32 const* sequence_lengths, SizeType32 max_seq_len, hipStream_t stream);

} // namespace kernels
} // namespace tensorrt_llm
