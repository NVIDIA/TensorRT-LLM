#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/stopCriteriaKernels.h"

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{
namespace kernels
{

namespace
{
__global__ void stopWordsCriterion(TokenIdType const** outputIds, SizeType32 const** parentIds,
    TokenIdType const* const* stopWords, FinishedState* finished, SizeType32* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 const* stopWordsLens, SizeType32* numNewTokens, SizeType32 batchSize,
    SizeType32 beamWidth, SizeType32 maxSeqLen)
{
    auto const id = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
    auto const batchIdx = blockIdx.y / beamWidth;
    auto const beamIdx = blockIdx.y % beamWidth;
    auto const batchSlot = batchSlots[batchIdx];
    auto const batchBeamIdx = batchSlot * beamWidth + beamIdx;
    auto const newTokens = numNewTokens ? numNewTokens[batchSlot] : 1;

    auto const* baseStopWords = stopWords[batchSlot];
    auto const stopWordsLen = stopWordsLens[batchSlot];
    auto const* baseOffsets = baseStopWords + stopWordsLen;

    if (id >= stopWordsLen || baseOffsets[id] < 0)
    {
        return;
    }

    auto const itemEnd = baseOffsets[id];
    auto const itemStart = (id > 0) ? baseOffsets[id - 1] : 0;
    auto const itemSize = itemEnd - itemStart;

    // The single-token case unconditionally bans the token
    bool shouldStop = false;
    SizeType32 stopLen = INT_MAX;
    SizeType32 step = 0;

    for (; step < newTokens; ++step)
    {
        // Need to minus newTokens because the sequenceLengths is already updated in this point
        auto const currentStep = sequenceLengths[batchBeamIdx] - newTokens + step;
        // Is sequence larger than stop word to look for a match?
        if (currentStep + 1 >= itemSize)
        {
            shouldStop = true;
            stopLen = currentStep + 1;
            auto parentId = static_cast<SizeType32>(beamIdx);
            bool const gatherBeam = beamWidth > 1;

            // Start from the last token
            for (auto tokenIdx = itemSize - 1; tokenIdx >= 0; tokenIdx--)
            {
                auto const previousToken
                    = outputIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];
                // If token does not match already, stop comparison
                if (previousToken != baseStopWords[itemStart + tokenIdx])
                {
                    shouldStop = false;
                    break;
                }
                if (gatherBeam)
                {
                    parentId = parentIds == nullptr
                        ? SizeType32{0}
                        : parentIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];

                    if (parentId < 0 || parentId >= beamWidth)
                    {
                        shouldStop = false;
                        break;
                    }
                }
            }
        }
        if (shouldStop)
        {
            finished[batchSlot * beamWidth + beamIdx].setFinishedStopWords();
            // When more than 1 token is predicted per step, find the first match with the stop word
            if (newTokens > 1)
            {
                // Update num of new tokens up to stopped word (including).
                atomicMin(numNewTokens + batchSlot, step + 1);
                // Update seq lengths up to stopped word (including).
                atomicMin(sequenceLengths + batchBeamIdx, stopLen);
            }
            break;
        }
    }
}
} // namespace

void invokeStopWordsCriterion(TokenIdType const** outputIds, SizeType32 const** parentIds,
    TokenIdType const* const* stopWords, FinishedState* finished, SizeType32* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 const* stopWordsLen, SizeType32* numNewTokens, SizeType32 maxStopWordsLen,
    SizeType32 batchSize, SizeType32 beamWidth, SizeType32 maxSeqLen, hipStream_t stream)
{
    // Check if we have sampled a word from the stopWords list. If so, stop the sequence.
    dim3 block, grid;
    constexpr SizeType32 maxBlockSize{256};

    block.x = min(((maxStopWordsLen + 32 - 1) / 32) * 32, maxBlockSize);
    grid.x = (maxStopWordsLen + block.x - 1) / block.x;
    grid.y = batchSize * beamWidth;

    stopWordsCriterion<<<grid, block, 0, stream>>>(outputIds, parentIds, stopWords, finished, sequenceLengths,
        batchSlots, stopWordsLen, numNewTokens, batchSize, beamWidth, maxSeqLen);
    sync_check_cuda_error(stream);
}

__global__ void lengthCriterion(FinishedState* finished, SizeType32* finishedSum, SizeType32 const* sequenceLimitLength,
    SizeType32* sequenceLengths, SizeType32* numNewTokens, SizeType32 const* batchSlots, SizeType32 batchSize,
    SizeType32 beamWidth)
{
    SizeType32 threadFinishedCount = 0;
    auto const batchIdx = blockIdx.x;
    auto const batchSlot = batchSlots[batchIdx];

    for (auto beamIdx = static_cast<SizeType32>(threadIdx.x); beamIdx < beamWidth;
         beamIdx += static_cast<SizeType32>(blockDim.x))
    {
        auto const batchSlotBeamWidthIdx = batchSlot * beamWidth + beamIdx;

        auto finishState = finished[batchSlotBeamWidthIdx];

        auto const numTokensToLimit = sequenceLimitLength[batchSlot] - sequenceLengths[batchSlotBeamWidthIdx];
        if (numTokensToLimit <= 0)
        {
            finishState.setFinishedMaxLength();
            sequenceLengths[batchSlotBeamWidthIdx] = sequenceLimitLength[batchSlot];
            if (numNewTokens)
            {
                numNewTokens[batchSlot] = numNewTokens[batchSlot] + numTokensToLimit;
            }
        }
        threadFinishedCount += finishState.isFinished() ? 1 : 0;
        finished[batchSlotBeamWidthIdx] = finishState;
    }

    if (finishedSum)
    {
        SizeType32 blockFinishedCount = 0;
        if (blockDim.x <= 32)
        {
            blockFinishedCount = warpReduceSum(threadFinishedCount);
        }
        else
        {
            blockFinishedCount = blockReduceSum(threadFinishedCount);
        }
        __syncthreads();

        if (threadIdx.x == 0)
        {
            finishedSum[batchSlot] = blockFinishedCount;
        }
    }
}

void invokeLengthCriterion(FinishedState* finished, SizeType32* finishedSum, SizeType32 const* sequenceLimitLength,
    SizeType32* sequenceLengths, SizeType32* numNewTokens, SizeType32 const* batchSlots, SizeType32 batchSize,
    SizeType32 beamWidth, hipStream_t stream)
{
    // Check if we have attained the sequence length limit. If so, stop the
    // sequence. In addition, check if all sequences are stopped and return the
    // result in shouldStop
    dim3 block{min(512, static_cast<uint32_t>(beamWidth))};
    dim3 grid{static_cast<uint32_t>(batchSize)};

    lengthCriterion<<<grid, block, 0, stream>>>(
        finished, finishedSum, sequenceLimitLength, sequenceLengths, numNewTokens, batchSlots, batchSize, beamWidth);
    sync_check_cuda_error(stream);
}

__global__ void explicitEOSCriterion(TokenIdType const** outputIds, TokenIdType const* endIds, FinishedState* finished,
    SizeType32* sequenceLengths, SizeType32* numNewTokens, SizeType32 const* batchSlots, SizeType32 batchSize,
    SizeType32 maxTokensPerStep)
{
    auto const batchIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (batchIdx >= batchSize)
    {
        return;
    }

    auto const batchSlot = batchSlots[batchIdx];
    if (finished[batchSlot].isFinished())
    {
        return;
    }

    auto const numTokens = numNewTokens != nullptr ? numNewTokens[batchSlot] : maxTokensPerStep;
    auto const endId = endIds[batchSlot];
    auto const sequenceLength = sequenceLengths[batchSlot];

    auto const posStart = max(0, sequenceLength - numTokens);
    auto const posEnd = sequenceLength;
    for (SizeType32 pos = posStart; pos < posEnd; ++pos)
    {
        auto const token = outputIds[batchSlot][pos];
        if (token == endId)
        {
            finished[batchSlot].setFinishedEOS();
            sequenceLengths[batchSlot] = max(0, pos);
            if (numNewTokens)
            {
                numNewTokens[batchSlot] = pos - posStart;
            }
            return;
        }
    }
}

void invokeExplicitEOSCriterion(TokenIdType const** outputIds, TokenIdType const* endIds, FinishedState* finished,
    SizeType32* sequenceLengths, SizeType32* numNewTokens, SizeType32 const* batchSlots, SizeType32 batchSize,
    SizeType32 beamWidth, SizeType32 maxTokensPerStep, hipStream_t stream)
{
    TLLM_CHECK_WITH_INFO(beamWidth == 1, "Explicit EOS criterion does not support beam search");
    // Check if we have sampled an end id token. If so, stop the sequence.
    SizeType32 constexpr blockSize{256};

    dim3 grid;
    grid.x = divUp(batchSize, blockSize);

    explicitEOSCriterion<<<grid, blockSize, 0, stream>>>(
        outputIds, endIds, finished, sequenceLengths, numNewTokens, batchSlots, batchSize, maxTokensPerStep);
    sync_check_cuda_error(stream);
}

} // namespace kernels
} // namespace tensorrt_llm
