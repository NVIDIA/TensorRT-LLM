#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cutlass_kernels/include/moe_kernels.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/kernels/cutlass_kernels/cutlass_type_conversion.h"
#include "tensorrt_llm/kernels/moeUtilOp.h"
#include "tensorrt_llm/kernels/quantization.cuh"

#include <hip/hip_fp16.h>
#include <float.h>

#include <climits> // For INT_MAX
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <cuda/std/limits> // For numeric_limits
#include <math.h>

#include <cutlass/array.h>
#include <cutlass/half.h>
#include <cutlass/numeric_types.h>

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#include <cub/device/device_radix_sort.cuh>
#include <cub/util_type.cuh>
#include <hiprand/hiprand_kernel.h>
#include <hiprand/hiprand_kernel.h>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#include "3rdparty/cub/device/device_radix_sort.cuh"
#include "3rdparty/cub/util_type.cuh"
#endif

namespace cg = cooperative_groups;
using namespace tensorrt_llm::common;

namespace tensorrt_llm::kernels
{

// ========================== CUB Sorting things ====================================
CubKeyValueSorter::CubKeyValueSorter()
    : num_experts_(0)
    , num_bits_(sizeof(int) * 8)
{
}

int CubKeyValueSorter::expertsToBits(int num_experts)
{
    // Max value we represent is V = num_experts + (num_experts - 1) = 2 * num_experts - 1
    // The maximum number of bits is therefore floor(log2(V)) + 1
    return static_cast<int>(log2(2 * num_experts - 1)) + 1;
}

CubKeyValueSorter::CubKeyValueSorter(int const num_experts)
    : num_experts_(num_experts)
    , num_bits_(expertsToBits(num_experts))
{
}

void CubKeyValueSorter::updateNumExperts(int const num_experts)
{
    num_experts_ = num_experts;
    num_bits_ = expertsToBits(num_experts);
}

size_t CubKeyValueSorter::getWorkspaceSize(size_t const num_key_value_pairs, int const num_experts)
{
    int num_bits = expertsToBits(num_experts);
    size_t required_storage = 0;
    int* null_int = nullptr;
    hipcub::DeviceRadixSort::SortPairs(
        nullptr, required_storage, null_int, null_int, null_int, null_int, num_key_value_pairs, 0, num_bits);

    // TODO: fix DeviceRadixSort
    //   when num_key_value_pairs, num_experts, num_bits, required_storage = 64, 4, 3, 0
    //   The required_storage seems to vary between 0 and 1 for the same inputs
    if (required_storage == 0)
    {
        required_storage = 1;
    }
    return required_storage;
}

void CubKeyValueSorter::run(void* workspace, size_t const workspace_size, int const* keys_in, int* keys_out,
    int const* values_in, int* values_out, size_t const num_key_value_pairs, hipStream_t stream)
{
    size_t expected_ws_size = getWorkspaceSize(num_key_value_pairs, num_experts_);
    size_t actual_ws_size = workspace_size;

    TLLM_CHECK_WITH_INFO(expected_ws_size <= workspace_size,
        "[CubKeyValueSorter::run] The allocated workspace is too small to run this problem.");
    hipcub::DeviceRadixSort::SortPairs(
        workspace, actual_ws_size, keys_in, keys_out, values_in, values_out, num_key_value_pairs, 0, num_bits_, stream);
}

// TODO: These kernel implementations are duplicated in moe_kernels.cu. They will be refactored later (tracked by
// https://jirasw.nvidia.com/browse/TRTLLM-708)
template <int BLOCK_SIZE, int EXPERTS_PER_TOKEN, int LOG2_NUM_EXPERTS>
__global__ void fusedBuildExpertMapsSortFirstTokenKernel(int const* const token_selected_experts,
    int* const unpermuted_token_selected_experts, int* const permuted_source_token_ids,
    int64_t* const expert_first_token_offset, int64_t const num_tokens, int const experts_per_token,
    int const start_expert, int const end_expert, int const num_experts_per_node)
{
    // Only using block wise collective so we can only have one block
    assert(gridDim.x == 1);

    assert(start_expert <= end_expert);
    assert(num_experts_per_node == (end_expert - start_expert));
    assert(end_expert <= num_experts_per_node);
    assert(num_experts_per_node <= (1 << LOG2_NUM_EXPERTS));

    int const token = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    bool is_valid_token = token < num_tokens;

    // This is the masked expert id for this token
    int local_token_selected_experts[EXPERTS_PER_TOKEN];
    // This is the final permuted rank of this token (ranked by selected expert)
    int local_token_permuted_indices[EXPERTS_PER_TOKEN];

    // Wait PDL before reading token_selected_experts
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

// build expert map
// we need to populate expert ids for all threads, even if there are
// fewer tokens
#pragma unroll
    for (int i = 0; i < EXPERTS_PER_TOKEN; i++)
    {
        int const expert
            = is_valid_token ? token_selected_experts[token * EXPERTS_PER_TOKEN + i] : num_experts_per_node;

        // If the token is not valid, set the expert id to num_experts_per_node + 1
        // If expert is not in the current node, set it to num_experts_per_node
        // If expert is in the current node, subtract start_expert to shift the range to [0, num_experts_per_node)
        bool is_valid_expert = expert >= start_expert && expert < end_expert;
        local_token_selected_experts[i] = !is_valid_token ? num_experts_per_node + 1
            : is_valid_expert                             ? (expert - start_expert)
                                                          : num_experts_per_node;
    }

    // TODO: decompose cub's sort to expose the bucket starts, and just return
    // that to elide the binary search

    // sort the expert map
    using BlockRadixRank = hipcub::BlockRadixRank<BLOCK_SIZE, LOG2_NUM_EXPERTS, false>;
    extern __shared__ unsigned char temp_storage[];
    auto& sort_temp = *reinterpret_cast<typename BlockRadixRank::TempStorage*>(temp_storage);

    // Sanity check that the number of bins do correspond to the number of experts
    static_assert(BlockRadixRank::BINS_TRACKED_PER_THREAD * BLOCK_SIZE >= (1 << LOG2_NUM_EXPERTS));
    assert(BlockRadixRank::BINS_TRACKED_PER_THREAD * BLOCK_SIZE >= num_experts_per_node);

    int local_expert_first_token_offset[BlockRadixRank::BINS_TRACKED_PER_THREAD];

    hipcub::BFEDigitExtractor<int> extractor(0, LOG2_NUM_EXPERTS);
    BlockRadixRank(sort_temp).RankKeys(
        local_token_selected_experts, local_token_permuted_indices, extractor, local_expert_first_token_offset);

// We are done with compute, launch the dependent kernels while the stores are in flight
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif

    // write to shared memory and global memory
    if (is_valid_token)
    {
#pragma unroll
        for (int i = 0; i < EXPERTS_PER_TOKEN; i++)
        {
            unpermuted_token_selected_experts[token * EXPERTS_PER_TOKEN + i] = local_token_selected_experts[i];
            permuted_source_token_ids[local_token_permuted_indices[i]] = i * num_tokens + token;
        }
    }

#pragma unroll
    for (int expert_id = 0; expert_id < BlockRadixRank::BINS_TRACKED_PER_THREAD; expert_id++)
    {
        int out_expert_id = expert_id + token * BlockRadixRank::BINS_TRACKED_PER_THREAD;
        if (out_expert_id < num_experts_per_node + 1)
        {
            expert_first_token_offset[out_expert_id] = local_expert_first_token_offset[expert_id];
        }
    }
}

template <int BLOCK_SIZE, int EXPERTS_PER_TOKEN, int LOG2_NUM_EXPERTS>
bool fusedBuildExpertMapsSortFirstTokenDispatch(int const* token_selected_experts,
    int* unpermuted_token_selected_experts, int* permuted_source_token_ids, int64_t* expert_first_token_offset,
    int64_t const num_tokens, int const num_experts_per_node, int const experts_per_token, int const start_expert,
    int const end_expert, hipStream_t stream)
{
    TLLM_CHECK_WITH_INFO(num_experts_per_node == (end_expert - start_expert),
        "num_experts_per_node must be equal to end_expert - start_expert");
    int const threads = BLOCK_SIZE;
    int const blocks = (num_tokens + threads - 1) / threads;
    TLLM_CHECK_WITH_INFO(blocks == 1, "Current implementation requires single block");

    using BlockRadixRank = hipcub::BlockRadixRank<BLOCK_SIZE, LOG2_NUM_EXPERTS, false>;
    size_t shared_size = sizeof(typename BlockRadixRank::TempStorage);

    hipLaunchConfig_t config;
    config.gridDim = blocks;
    config.blockDim = threads;
    config.dynamicSmemBytes = shared_size;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;

    auto kernel = &fusedBuildExpertMapsSortFirstTokenKernel<BLOCK_SIZE, EXPERTS_PER_TOKEN, LOG2_NUM_EXPERTS>;

    int device = 0;
    int max_smem_per_block = 0;
    check_cuda_error(hipGetDevice(&device));
    check_cuda_error(hipDeviceGetAttribute(&max_smem_per_block, hipDeviceAttributeSharedMemPerBlockOptin, device));
    if (shared_size >= static_cast<size_t>(max_smem_per_block))
    {
        // This should mean that
        // hipFuncSetAttribute(reinterpret_cast<const void*>(cutlass::Kernel<GemmKernel>), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size)
        // wouldn't work.
        return false;
    }

    check_cuda_error(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_size));
    check_cuda_error(cudaLaunchKernelEx(&config, kernel, token_selected_experts, unpermuted_token_selected_experts,
        permuted_source_token_ids, expert_first_token_offset, num_tokens, experts_per_token, start_expert, end_expert,
        num_experts_per_node));

    return true;
}

template <int EXPERTS_PER_TOKEN, int LOG2_NUM_EXPERTS>
bool fusedBuildExpertMapsSortFirstTokenBlockSize(int const* token_selected_experts,
    int* unpermuted_token_selected_experts, int* permuted_source_token_ids, int64_t* expert_first_token_offset,
    int64_t const num_tokens, int const num_experts_per_node, int const experts_per_token, int const start_expert,
    int const end_expert, hipStream_t stream)
{
    int const block_size = num_tokens;
    if (num_tokens > 256)
    {
        TLLM_LOG_TRACE(
            "Number of tokens %d is greater than 256, which is not supported for fused moe prologues", num_tokens);
        return false;
    }

    auto func = &fusedBuildExpertMapsSortFirstTokenDispatch<32, EXPERTS_PER_TOKEN, LOG2_NUM_EXPERTS>;
    if (block_size > 32 && block_size <= 64)
    {
        func = &fusedBuildExpertMapsSortFirstTokenDispatch<64, EXPERTS_PER_TOKEN, LOG2_NUM_EXPERTS>;
    }
    else if (block_size > 64 && block_size <= 128)
    {
        func = &fusedBuildExpertMapsSortFirstTokenDispatch<128, EXPERTS_PER_TOKEN, LOG2_NUM_EXPERTS>;
    }
    else if (block_size > 128 && block_size <= 256)
    {
        func = &fusedBuildExpertMapsSortFirstTokenDispatch<256, EXPERTS_PER_TOKEN, LOG2_NUM_EXPERTS>;
    }

    return func(token_selected_experts, unpermuted_token_selected_experts, permuted_source_token_ids,
        expert_first_token_offset, num_tokens, num_experts_per_node, experts_per_token, start_expert, end_expert,
        stream);
}

template <int LOG2_NUM_EXPERTS>
bool fusedBuildExpertMapsSortFirstTokenBlockSize(int const* token_selected_experts,
    int* unpermuted_token_selected_experts, int* permuted_source_token_ids, int64_t* expert_first_token_offset,
    int64_t const num_tokens, int const num_experts_per_node, int const experts_per_token, int const start_expert,
    int const end_expert, hipStream_t stream)
{
    auto func = &fusedBuildExpertMapsSortFirstTokenBlockSize<1, LOG2_NUM_EXPERTS>;
    switch (experts_per_token)
    {
    case 1:
    {
        func = &fusedBuildExpertMapsSortFirstTokenBlockSize<1, LOG2_NUM_EXPERTS>;
        break;
    }
    case 2:
    {
        func = &fusedBuildExpertMapsSortFirstTokenBlockSize<2, LOG2_NUM_EXPERTS>;
        break;
    }
    case 4:
    {
        func = &fusedBuildExpertMapsSortFirstTokenBlockSize<4, LOG2_NUM_EXPERTS>;
        break;
    }
    case 6:
    {
        func = &fusedBuildExpertMapsSortFirstTokenBlockSize<6, LOG2_NUM_EXPERTS>;
        break;
    }
    case 8:
    {
        func = &fusedBuildExpertMapsSortFirstTokenBlockSize<8, LOG2_NUM_EXPERTS>;
        break;
    }
    default:
    {
        TLLM_LOG_TRACE("Top-K value %d does not have supported fused moe prologues", experts_per_token);
        return false;
    }
    }
    return func(token_selected_experts, unpermuted_token_selected_experts, permuted_source_token_ids,
        expert_first_token_offset, num_tokens, num_experts_per_node, experts_per_token, start_expert, end_expert,
        stream);
}

bool fusedBuildExpertMapsSortFirstToken(int const* token_selected_experts, int* unpermuted_token_selected_experts,
    int* permuted_source_token_ids, int64_t* expert_first_token_offset, int64_t const num_tokens,
    int const num_experts_per_node, int const experts_per_token, int const start_expert, int const end_expert,
    hipStream_t stream)
{
    // We need enough bits to represent [0, num_experts_per_node+1] (inclusive) i.e. num_experts_per_node + 2 values
    // This is floor(log2(num_experts_per_node+1)) + 1
    int expert_log = static_cast<int>(log2(num_experts_per_node + 1)) + 1;
    if (expert_log <= 9)
    {
        auto funcs = std::array{&fusedBuildExpertMapsSortFirstTokenBlockSize<1>,
            &fusedBuildExpertMapsSortFirstTokenBlockSize<2>, &fusedBuildExpertMapsSortFirstTokenBlockSize<3>,
            &fusedBuildExpertMapsSortFirstTokenBlockSize<4>, &fusedBuildExpertMapsSortFirstTokenBlockSize<5>,
            &fusedBuildExpertMapsSortFirstTokenBlockSize<6>, &fusedBuildExpertMapsSortFirstTokenBlockSize<7>,
            &fusedBuildExpertMapsSortFirstTokenBlockSize<8>, &fusedBuildExpertMapsSortFirstTokenBlockSize<9>};

        return funcs[expert_log - 1](token_selected_experts, unpermuted_token_selected_experts,
            permuted_source_token_ids, expert_first_token_offset, num_tokens, num_experts_per_node, experts_per_token,
            start_expert, end_expert, stream);
    }
    TLLM_LOG_TRACE("Experts per node %d does not have supported fused moe prologues", num_experts_per_node);
    return false;
}

// ============================== Infer GEMM sizes =================================
// TODO Could linear search be better for small # experts
template <class T>
__device__ inline int64_t findTotalEltsLessThanTarget(T const* sorted_indices, int64_t const arr_length, T const target)
{
    int64_t low = 0, high = arr_length - 1, target_location = -1;
    while (low <= high)
    {
        int64_t mid = (low + high) / 2;

        if (sorted_indices[mid] >= target)
        {
            high = mid - 1;
        }
        else
        {
            low = mid + 1;
            target_location = mid;
        }
    }
    return target_location + 1;
}

// Calculates the start offset of the tokens for a given expert. The last element is the total number of valid tokens
__global__ void computeExpertFirstTokenOffsetKernel(int const* sorted_experts, int64_t const sorted_experts_len,
    int64_t const num_experts_per_node, int64_t* expert_first_token_offset)
{
    // First, compute the global tid. We only need 1 thread per expert.
    int const expert = blockIdx.x * blockDim.x + threadIdx.x;

    // Note that expert goes [0, num_experts] (inclusive) because we want a count for the total number of active tokens
    // at the end of the scan.
    if (expert >= num_experts_per_node + 1)
    {
        return;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif
    expert_first_token_offset[expert] = findTotalEltsLessThanTarget(sorted_experts, sorted_experts_len, expert);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

void computeExpertFirstTokenOffset(int const* sorted_indices, int const total_indices, int const num_experts_per_node,
    int64_t* expert_first_token_offset, hipStream_t stream)
{
    int const num_entries = num_experts_per_node + 1;
    int const threads = std::min(1024, num_entries);
    int const blocks = (num_entries + threads - 1) / threads;

    hipLaunchConfig_t config;
    config.gridDim = blocks;
    config.blockDim = threads;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    cudaLaunchKernelEx(&config, computeExpertFirstTokenOffsetKernel, sorted_indices, total_indices,
        num_experts_per_node, expert_first_token_offset);
}

template <class T>
using sizeof_bits = cutlass::sizeof_bits<typename cutlass_kernels::TllmToCutlassTypeAdapter<std::remove_cv_t<T>>::type>;

// Function to safely offset an pointer that may contain sub-byte types (FP4/INT4)
template <class T>
__host__ __device__ constexpr T* safe_inc_ptr(T* ptr, size_t offset)
{
    constexpr int adjustment = (sizeof_bits<T>::value < 8) ? (8 / sizeof_bits<T>::value) : 1;
    assert(offset % adjustment == 0 && "Attempt to offset index to sub-byte");
    return ptr + offset / adjustment;
}

__host__ __device__ constexpr int64_t getOffsetActivationSF(int64_t expert_id, int64_t token_offset, int64_t gemm_k,
    cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::FpXBlockScalingType scaling_type)
{
    auto function = [=](int64_t min_alignment, int64_t block_size)
    {
        // This formulation ensures that sf_offset[i + 1] - sf_offset[i] >= token_offset[i + 1] - token_offset[i].
        int64_t sf_offset = (token_offset + expert_id * (min_alignment - 1)) / min_alignment * min_alignment;
        assert(gemm_k % block_size == 0);
        return sf_offset * gemm_k / block_size;
    };
    switch (scaling_type)
    {
    case cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::FpXBlockScalingType::MXFPX:
        return function(cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::MinNumRowsAlignmentMXFPX,
            cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::MXFPXBlockScaleVectorSize);
    case cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::FpXBlockScalingType::NVFP4:
        return function(cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::MinNumRowsAlignmentNVFP4,
            cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::NVFP4BlockScaleVectorSize);
    case cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::FpXBlockScalingType::NONE:
        return 0; // No scaling factors, no offset
    }

    assert(false && "Unrecognized scaling type");
    return 0;
}

constexpr static int NVFP4_VEC_SIZE = 16;

template <class GemmOutputType, class ComputeElem>
__device__ uint32_t quantizePackedFP4Value(ComputeElem& post_act_val, float global_scale_val,
    int64_t num_tokens_before_expert, int64_t expert_id, int64_t token_id, int64_t elem_idx, int64_t num_cols,
    int64_t max_tokens_per_expert, cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF* act_sf_flat,
    cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::FpXBlockScalingType scaling_type)
{
    static constexpr int CVT_FP4_NUM_THREADS_PER_SF = NVFP4_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD;
    // Quantize the input to FP4
    static_assert(std::is_same_v<GemmOutputType, __hip_bfloat16> || std::is_same_v<GemmOutputType, half>);
    static_assert(ComputeElem::kElements == CVT_FP4_ELTS_PER_THREAD);
    PackedVec<GemmOutputType> packed_vec{};
    for (int i = 0; i < CVT_FP4_ELTS_PER_THREAD / 2; i++)
    {
        packed_vec.elts[i].x = static_cast<GemmOutputType>(post_act_val[i * 2 + 0]);
        packed_vec.elts[i].y = static_cast<GemmOutputType>(post_act_val[i * 2 + 1]);
    }

    // We need to offset into the scaling factors for just this expert
    auto act_sf_expert
        = act_sf_flat + getOffsetActivationSF(expert_id, num_tokens_before_expert, num_cols, scaling_type);

    // Use `token - num_tokens_before_expert` because we want this to be relative to the start of this expert
    auto sf_out = cvt_quant_to_fp4_get_sf_out_offset<cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF,
        CVT_FP4_NUM_THREADS_PER_SF, NVFP4_VEC_SIZE>(std::nullopt /* batchIdx */, token_id - num_tokens_before_expert,
        elem_idx, std::nullopt /* numRows */, num_cols, act_sf_expert, FP4QuantizationSFLayout::SWIZZLED);

    // Do the conversion and set the output and scaling factor
    auto func = (scaling_type == cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::FpXBlockScalingType::NVFP4)
        ? &cvt_warp_fp16_to_fp4<GemmOutputType, NVFP4_VEC_SIZE, false>
        : &cvt_warp_fp16_to_fp4<GemmOutputType, NVFP4_VEC_SIZE, true>;
    auto res = func(packed_vec, global_scale_val, sf_out);
    return res;
}

__device__ void writeSF(int64_t num_tokens_before_expert, int64_t expert_id, int64_t source_token_id, int64_t token_id,
    int64_t elem_idx, int64_t num_cols, int64_t max_tokens_per_expert,
    cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF* act_sf_flat,
    cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF const* input_sf)
{
    static constexpr int CVT_FP4_NUM_THREADS_PER_SF = NVFP4_VEC_SIZE / CVT_FP4_ELTS_PER_THREAD;

    // We need to offset into the scaling factors for just this expert
    auto act_sf_expert = act_sf_flat
        + getOffsetActivationSF(expert_id, num_tokens_before_expert, num_cols,
            cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::FpXBlockScalingType::NVFP4);

    // Use `token - num_tokens_before_expert` because we want this to be relative to the start of this expert
    auto sf_out = cvt_quant_to_fp4_get_sf_out_offset<cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF,
        CVT_FP4_NUM_THREADS_PER_SF, NVFP4_VEC_SIZE>(std::nullopt /* batchIdx */, token_id - num_tokens_before_expert,
        elem_idx, std::nullopt /* numRows */, num_cols, act_sf_expert, FP4QuantizationSFLayout::SWIZZLED);
    if (sf_out)
    {
        auto const sf_in
            = cvt_quant_to_fp4_get_sf_out_offset<cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF,
                CVT_FP4_NUM_THREADS_PER_SF, NVFP4_VEC_SIZE>(std::nullopt /* batchIdx */, source_token_id, elem_idx,
                std::nullopt /* numRows */, num_cols,
                const_cast<cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF*>(input_sf),
                FP4QuantizationSFLayout::SWIZZLED);
        *sf_out = *sf_in;
    }
}

void generateTokenPermutation(int const* unpermuted_token_selected_experts, int const* unpermuted_source_token_ids,
    int* permuted_token_selected_experts, int* permuted_source_token_ids, int64_t* expert_first_token_offset,
    int64_t num_rows, int64_t num_experts_per_node, int64_t k, CubKeyValueSorter& sorter, void* sorter_ws,
    hipStream_t stream)
{
    int64_t const expanded_num_rows = k * num_rows;
    sorter.updateNumExperts(num_experts_per_node);
    size_t const sorter_ws_size_bytes
        = pad_to_multiple_of_16(sorter.getWorkspaceSize(expanded_num_rows, num_experts_per_node));
    sorter.run((void*) sorter_ws, sorter_ws_size_bytes, unpermuted_token_selected_experts,
        permuted_token_selected_experts, unpermuted_source_token_ids, permuted_source_token_ids, expanded_num_rows,
        stream);

    sync_check_cuda_error(stream);

    // Upper bound on number of expanded rows
    computeExpertFirstTokenOffset(
        permuted_token_selected_experts, expanded_num_rows, num_experts_per_node, expert_first_token_offset, stream);
}

/**
 * Takes the input maps and prepares the expanded maps for the sort step
 * @param unpermuted_token_selected_experts: Buffer of transformed expert ids masked for the current node, used as the
 * keys for the sort
 * @param unpermuted_source_token_ids: Buffer of unpermuted token ids that will be used to identify the source row for
 * each expanded token, used as the values for the sort
 */
__global__ void buildExpertMapsKernel(int const* token_selected_experts, int* unpermuted_token_selected_experts,
    int* unpermuted_source_token_ids, int64_t const num_tokens, int const experts_per_token, int const start_expert,
    int const end_expert, int const num_experts_per_node)
{
    int const token = blockIdx.x * blockDim.x + threadIdx.x;
    if (token >= num_tokens)
    {
        return;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

    for (int i = 0; i < experts_per_token; i++)
    {
        int const expert = token_selected_experts[token * experts_per_token + i];
        // If expert is not in the current node, set it to num_experts_per_node
        // If expert is in the current node, subtract start_expert to shift the range to [0, num_experts_per_node)
        bool is_valid_expert = expert >= start_expert && expert < end_expert;
        unpermuted_token_selected_experts[token * experts_per_token + i]
            = is_valid_expert ? (expert - start_expert) : num_experts_per_node;
        unpermuted_source_token_ids[token * experts_per_token + i] = i * num_tokens + token;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

void buildExpertMaps(int const* token_selected_experts, int* unpermuted_token_selected_experts,
    int* unpermuted_source_token_ids, int64_t const num_tokens, int const num_experts_per_node,
    int const experts_per_token, int const start_expert, int const end_expert, hipStream_t stream)
{
    TLLM_CHECK_WITH_INFO(num_experts_per_node == (end_expert - start_expert),
        "num_experts_per_node must be equal to end_expert - start_expert");
    int const threads = std::min(int64_t(1024), num_tokens);
    int const blocks = (num_tokens + threads - 1) / threads;

    hipLaunchConfig_t config;
    config.gridDim = blocks;
    config.blockDim = threads;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    cudaLaunchKernelEx(&config, buildExpertMapsKernel, token_selected_experts, unpermuted_token_selected_experts,
        unpermuted_source_token_ids, num_tokens, experts_per_token, start_expert, end_expert, num_experts_per_node);
}

// ========================== Permutation things =======================================
template <class T, class U>
__host__ __device__ constexpr static U arrayConvert(T const& input)
{
    using Type = typename U::Element;
    static_assert(T::kElements == U::kElements);
    U u;
#pragma unroll
    for (int i = 0; i < U::kElements; i++)
    {
        u[i] = static_cast<Type>(input[i]);
    }
    return u;
}

// Duplicated and permutes rows for MoE. In addition, reverse the permutation map to help with finalizing routing.

// "expanded_x_row" simply means that the number of values is num_rows x k. It is "expanded" since we will have to
// duplicate some rows in the input matrix to match the dimensions. Duplicates will always get routed to separate
// experts in the end.

// Note that the expanded_dest_row_to_expanded_source_row map referred to here has indices in the range (0,
// k*rows_in_input - 1). However, it is set up so that index 0, rows_in_input, 2*rows_in_input ... (k-1)*rows_in_input
// all map to row 0 in the original matrix. Thus, to know where to read in the source matrix, we simply take the modulus
// of the expanded index.

constexpr static int EXPAND_THREADS_PER_BLOCK = 256;

template <class InputActivationsType, class ExpandedActivationsType, bool CHECK_SKIPPED>
__global__ void expandInputRowsKernel(InputActivationsType const* unpermuted_input,
    ExpandedActivationsType* permuted_output, float const* unpermuted_scales, float* permuted_scales,
    int const* expanded_dest_row_to_expanded_source_row, int* expanded_source_row_to_expanded_dest_row,
    int64_t const num_rows, int64_t const* num_dest_rows, int64_t const cols, int64_t k,
    float const* fc1_act_global_scale, int64_t* expert_first_token_offset,
    cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF* fc1_act_sf_flat,
    cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF const* input_sf, int64_t num_experts_per_node)
{
#ifdef ENABLE_FP4
    constexpr bool is_fp4 = std::is_same_v<ExpandedActivationsType, __hip_fp4_e2m1>;
    constexpr bool is_fp4_input = is_fp4 && std::is_same_v<InputActivationsType, __hip_fp4_e2m1>;
    constexpr bool need_fp4_quant = is_fp4 && !std::is_same_v<InputActivationsType, __hip_fp4_e2m1>;
#else
    constexpr bool is_fp4 = false;
    constexpr bool is_fp4_input = false;
    constexpr bool need_fp4_quant = false;
#endif

    static_assert(need_fp4_quant || std::is_same_v<InputActivationsType, ExpandedActivationsType>,
        "Only FP4 quantization supports outputting a different format as part of the expansion");

    // Reverse permutation map.
    // I do this so that later, we can use the source -> dest map to do the k-way reduction and unpermuting. I need the
    // reverse map for that reduction to allow each threadblock to do 1 k-way reduce without atomics later in MoE. 1
    // thread block will be responsible for all k summations.
    int64_t const expanded_dest_row = blockIdx.x;
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif
    int64_t const expanded_source_row = expanded_dest_row_to_expanded_source_row[expanded_dest_row];
    if (threadIdx.x == 0)
    {
        assert(expanded_dest_row <= INT32_MAX);
        expanded_source_row_to_expanded_dest_row[expanded_source_row] = static_cast<int>(expanded_dest_row);
    }

    if (!CHECK_SKIPPED || blockIdx.x < *num_dest_rows)
    {
        // Load 128-bits per thread
        constexpr int64_t ELEM_PER_THREAD
            = is_fp4 ? CVT_FP4_ELTS_PER_THREAD : (128 / sizeof_bits<InputActivationsType>::value);
        constexpr int64_t ELEM_PER_BYTE = is_fp4_input ? 2 : 1;
        using DataElem
            = std::conditional_t<is_fp4_input, uint32_t, cutlass::Array<InputActivationsType, ELEM_PER_THREAD>>;
        using OutputElem = std::conditional_t<is_fp4, uint32_t, DataElem>;

        // Duplicate and permute rows
        int64_t const source_k_rank = expanded_source_row / num_rows;
        int64_t const source_row = expanded_source_row % num_rows;

        auto const* source_row_ptr
            = reinterpret_cast<DataElem const*>(unpermuted_input + source_row * cols / ELEM_PER_BYTE);
        // Cast first to handle when this is FP4
        auto* dest_row_ptr
            = reinterpret_cast<OutputElem*>(permuted_output) + expanded_dest_row * cols / ELEM_PER_THREAD;

        int64_t const start_offset = threadIdx.x;
        int64_t const stride = EXPAND_THREADS_PER_BLOCK;
        int64_t const num_elems_in_col = cols / ELEM_PER_THREAD;
        assert(cols % ELEM_PER_THREAD == 0);

        if constexpr (is_fp4)
        {
            int64_t expert = findTotalEltsLessThanTarget(
                                 expert_first_token_offset, num_experts_per_node, (int64_t) expanded_dest_row + 1)
                - 1;
            float global_scale_val = fc1_act_global_scale ? *fc1_act_global_scale : 1.0f;
            int64_t num_tokens_before_expert = expert_first_token_offset[expert];

            for (int elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride)
            {
                auto in_vec = source_row_ptr[elem_index];
                if constexpr (need_fp4_quant)
                {
                    // auto res = quantizePackedFP4Value<InputActivationsType, DataElem>(in_vec, global_scale_val,
                    //     num_tokens_before_expert, expert, expanded_dest_row, elem_index, cols, num_rows,
                    //     fc1_act_sf_flat);
                    auto res = quantizePackedFP4Value<InputActivationsType, DataElem>(in_vec, global_scale_val,
                        num_tokens_before_expert, expert, expanded_dest_row, elem_index, cols, num_rows,
                        fc1_act_sf_flat,
                        cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::FpXBlockScalingType::NVFP4);
                    dest_row_ptr[elem_index] = res;
                }
                else
                {
                    writeSF(num_tokens_before_expert, expert, source_row, expanded_dest_row, elem_index, cols, num_rows,
                        fc1_act_sf_flat, input_sf);
                    dest_row_ptr[elem_index] = in_vec;
                }
            }
        }
        else
        {
            for (int elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride)
            {
                dest_row_ptr[elem_index] = source_row_ptr[elem_index];
            }
        }

        if (permuted_scales && threadIdx.x == 0)
        {
            int64_t const source_k_idx = source_row * k + source_k_rank;
            permuted_scales[expanded_dest_row] = unpermuted_scales ? unpermuted_scales[source_k_idx] : 1.0f;
        }
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <class InputActivationsType, class ExpandedActivationsType>
void expandInputRowsKernelLauncher(InputActivationsType const* unpermuted_input,
    ExpandedActivationsType* permuted_output, float const* unpermuted_scales, float* permuted_scales,
    int const* expanded_dest_row_to_expanded_source_row, int* expanded_source_row_to_expanded_dest_row,
    int64_t const num_rows, int64_t const* num_valid_tokens_ptr, int64_t const cols, int const k,
    int const num_experts_per_node, float const* fc1_act_global_scale, int64_t* expert_first_token_offset,
    cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF* fc1_act_sf_flat,
    cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF const* input_sf, hipStream_t stream)
{
    if (fc1_act_sf_flat)
    {
        assert(false && "Not supported, we need to keep the same as moe_kerenls.cu in the future (TODO).");
    }

    int64_t const blocks = num_rows * k;
    int64_t const threads = EXPAND_THREADS_PER_BLOCK;
    auto func = (num_valid_tokens_ptr != nullptr)
        ? expandInputRowsKernel<InputActivationsType, ExpandedActivationsType, true>
        : expandInputRowsKernel<InputActivationsType, ExpandedActivationsType, false>;

    hipLaunchConfig_t config;
    config.gridDim = blocks;
    config.blockDim = threads;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    cudaLaunchKernelEx(&config, func, unpermuted_input, permuted_output, unpermuted_scales, permuted_scales,
        expanded_dest_row_to_expanded_source_row, expanded_source_row_to_expanded_dest_row, num_rows,
        num_valid_tokens_ptr, cols, k, fc1_act_global_scale, expert_first_token_offset, fc1_act_sf_flat, input_sf,
        num_experts_per_node);
}

#define INSTANTIATE_EXPAND_INPUT_ROWS(InputActivationsType, ExpandedActivationsType)                                   \
    template void expandInputRowsKernelLauncher<InputActivationsType, ExpandedActivationsType>(                        \
        InputActivationsType const* unpermuted_input, ExpandedActivationsType* permuted_output,                        \
        float const* unpermuted_scales, float* permuted_scales, int const* expanded_dest_row_to_expanded_source_row,   \
        int* expanded_source_row_to_expanded_dest_row, int64_t const num_rows, int64_t const* num_valid_tokens_ptr,    \
        int64_t const cols, int const k, int const num_experts_per_node, float const* fc1_act_global_scale,            \
        int64_t* expert_first_token_offset,                                                                            \
        cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF* fc1_act_sf_flat,                               \
        cutlass_kernels::TmaWarpSpecializedGroupedGemmInput::ElementSF const* input_sf, hipStream_t stream);

INSTANTIATE_EXPAND_INPUT_ROWS(half, half);
INSTANTIATE_EXPAND_INPUT_ROWS(float, float);
#ifdef ENABLE_BF16
INSTANTIATE_EXPAND_INPUT_ROWS(__hip_bfloat16, __hip_bfloat16);
#endif

enum class ScaleMode : int
{
    NO_SCALE = 0,
    DEFAULT = 1,
};

constexpr static int FINALIZE_THREADS_PER_BLOCK = 256;

template <class T>
using sizeof_bits = cutlass::sizeof_bits<typename cutlass_kernels::TllmToCutlassTypeAdapter<std::remove_cv_t<T>>::type>;

// Final kernel to unpermute and scale
// This kernel unpermutes the original data, does the k-way reduction and performs the final skip connection.
template <typename OutputType, class GemmOutputType, class ScaleBiasType, ScaleMode SCALE_MODE, bool CHECK_SKIPPED>
__global__ void finalizeMoeRoutingKernel(GemmOutputType const* expanded_permuted_rows,
    OutputType* reduced_unpermuted_output, ScaleBiasType const* bias, float const* scales,
    int const* expanded_source_row_to_expanded_dest_row, int const* expert_for_source_row, int64_t const orig_cols,
    int64_t const experts_per_token, int64_t const* num_valid_ptr)
{
    assert(orig_cols % 4 == 0);
    int64_t const original_row = blockIdx.x;
    int64_t const num_rows = gridDim.x;
    auto const offset = original_row * orig_cols;
    OutputType* reduced_row_ptr = reduced_unpermuted_output + offset;

    // Load 128-bits per thread, according to the smallest data type we read/write
    constexpr int64_t FINALIZE_ELEM_PER_THREAD
        = 128 / std::min(sizeof_bits<OutputType>::value, sizeof_bits<GemmOutputType>::value);

    int64_t const start_offset = threadIdx.x;
    int64_t const stride = FINALIZE_THREADS_PER_BLOCK;
    int64_t const num_elems_in_col = orig_cols / FINALIZE_ELEM_PER_THREAD;

    using BiasElem = cutlass::Array<ScaleBiasType, FINALIZE_ELEM_PER_THREAD>;
    using InputElem = cutlass::Array<GemmOutputType, FINALIZE_ELEM_PER_THREAD>;
    using OutputElem = cutlass::Array<OutputType, FINALIZE_ELEM_PER_THREAD>;
    using ComputeElem = cutlass::Array<float, FINALIZE_ELEM_PER_THREAD>;
    auto const* bias_v = reinterpret_cast<BiasElem const*>(bias);
    auto const* expanded_permuted_rows_v = reinterpret_cast<InputElem const*>(expanded_permuted_rows);
    auto* reduced_row_ptr_v = reinterpret_cast<OutputElem*>(reduced_row_ptr);

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif
    int64_t const num_valid = *num_valid_ptr;

#pragma unroll
    for (int elem_index = start_offset; elem_index < num_elems_in_col; elem_index += stride)
    {
        bool has_valid = false;
        ComputeElem thread_output;
        thread_output.fill(0);
        for (int k_idx = 0; k_idx < experts_per_token; ++k_idx)
        {
            int64_t const expanded_original_row = original_row + k_idx * num_rows;
            int64_t const expanded_permuted_row = expanded_source_row_to_expanded_dest_row[expanded_original_row];

            int64_t const k_offset = original_row * experts_per_token + k_idx;
            float const row_scale = (SCALE_MODE == ScaleMode::NO_SCALE) ? 1.f : scales[k_offset];

            // Check after row_rescale has accumulated
            if (CHECK_SKIPPED && expanded_permuted_row >= num_valid)
            {
                continue;
            }

            auto const* expanded_permuted_rows_row_ptr
                = expanded_permuted_rows_v + expanded_permuted_row * num_elems_in_col;

            int64_t const expert_idx = expert_for_source_row[k_offset];

            auto const* bias_ptr = bias_v + expert_idx * num_elems_in_col;
            ComputeElem bias_value;
            if (bias)
            {
                bias_value = arrayConvert<BiasElem, ComputeElem>(bias_ptr[elem_index]);
            }
            else
            {
                bias_value.fill(0);
            }

            ComputeElem expert_result
                = arrayConvert<InputElem, ComputeElem>(expanded_permuted_rows_row_ptr[elem_index]);
            thread_output = thread_output + row_scale * (expert_result + bias_value);
            has_valid = true;
        }

        OutputElem output_elem = arrayConvert<ComputeElem, OutputElem>(thread_output);
        reduced_row_ptr_v[elem_index] = output_elem;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <class OutputType, class GemmOutputType, class ScaleBiasType>
void finalizeMoeRoutingKernelLauncher(GemmOutputType const* expanded_permuted_rows,
    OutputType* reduced_unpermuted_output, ScaleBiasType const* bias, float const* final_scales,
    int const* expanded_source_row_to_expanded_dest_row, int const* expert_for_source_row, int64_t const num_rows,
    int64_t const cols, int64_t const experts_per_token, int64_t const* num_valid_ptr,
    cutlass_kernels::MOEParallelismConfig parallelism_config, hipStream_t stream)
{
    int64_t const blocks = num_rows;
    int64_t const threads = FINALIZE_THREADS_PER_BLOCK;

    // Only add bias on rank 0 for tensor parallelism
    bool const is_rank_0 = parallelism_config.tp_rank == 0;
    ScaleBiasType const* bias_ptr = is_rank_0 ? bias : nullptr;

    bool const check_skipped = num_valid_ptr != nullptr;

    ScaleMode scale_mode = final_scales ? ScaleMode::DEFAULT : ScaleMode::NO_SCALE;

    using FuncPtr
        = decltype(&finalizeMoeRoutingKernel<OutputType, GemmOutputType, ScaleBiasType, ScaleMode::DEFAULT, false>);
    FuncPtr func_map[2][3] = {
        {
            &finalizeMoeRoutingKernel<OutputType, GemmOutputType, ScaleBiasType, ScaleMode::NO_SCALE, false>,
            &finalizeMoeRoutingKernel<OutputType, GemmOutputType, ScaleBiasType, ScaleMode::DEFAULT, false>,
        },
        {
            &finalizeMoeRoutingKernel<OutputType, GemmOutputType, ScaleBiasType, ScaleMode::NO_SCALE, true>,
            &finalizeMoeRoutingKernel<OutputType, GemmOutputType, ScaleBiasType, ScaleMode::DEFAULT, true>,
        },
    };
    auto* const func = func_map[check_skipped][int(scale_mode)];

    hipLaunchConfig_t config;
    config.gridDim = blocks;
    config.blockDim = threads;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    cudaLaunchKernelEx(&config, func, expanded_permuted_rows, reduced_unpermuted_output, bias_ptr, final_scales,
        expanded_source_row_to_expanded_dest_row, expert_for_source_row, cols, experts_per_token, num_valid_ptr);
}

#define INSTANTIATE_FINALIZE_MOE_ROUTING(OutputT, GemmOutputT, ScaleBiasT)                                             \
    template void finalizeMoeRoutingKernelLauncher<OutputT, GemmOutputT, ScaleBiasT>(                                  \
        GemmOutputT const* expanded_permuted_rows, OutputT* reduced_unpermuted_output, ScaleBiasT const* bias,         \
        float const* final_scales, int const* expanded_source_row_to_expanded_dest_row,                                \
        int const* expert_for_source_row, int64_t const num_rows, int64_t const cols, int64_t const experts_per_token, \
        int64_t const* num_valid_ptr, cutlass_kernels::MOEParallelismConfig parallelism_config, hipStream_t stream);

INSTANTIATE_FINALIZE_MOE_ROUTING(half, half, half);
INSTANTIATE_FINALIZE_MOE_ROUTING(float, float, float);
#ifdef ENABLE_BF16
INSTANTIATE_FINALIZE_MOE_ROUTING(__hip_bfloat16, __hip_bfloat16, __hip_bfloat16);
#endif

} // namespace tensorrt_llm::kernels
