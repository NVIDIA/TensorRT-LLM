#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaBf16Wrapper.h"
#include "tensorrt_llm/common/cudaUtils.h"

// TODO(oargov): literally zero performance optimization work was put into these kernels and their launch parameters,
// since they should hopefully be fused to some gemm eventually.
namespace tensorrt_llm::kernels
{
template <typename T>
__global__ void tokenPerChannelScaleKernel(size_t const numModules, size_t const numTokens,
    int64_t const* __restrict__ cumModuleSizes, T const* __restrict__ a, T const* const* __restrict__ scales,
    T* __restrict__ result)
{
    /*
     * This kernel applies DoRA scaling to LoRA output.
     * Like LoRA, each token in the batch may target a different adapter.
     * Each adapter may also have multiple modules, for example: QKV projection will have a different scale for Q, K and
     * V, but they will be concatenated into a single input vector.
     * `scales` is a vector of pointers to DoRA magnitude vectors. Each token will have `numModules` pointers, and
     * pointers for the same module are next to each other. For example:
     * scales = [token0_module0_ptr, token1_module0_ptr, ..., token0_module1_ptr, token1_module1_ptr, ...]
     */
    auto const threadId = blockIdx.x * blockDim.x + threadIdx.x;
    // number of columns in the input
    auto const numChannels = cumModuleSizes[numModules - 1];
    // current thread's column
    auto const channelId = threadId % numChannels;
    // current thread's token
    auto const tokenId = threadId / numChannels;
    // offset the input column to fit in the scaling vector's column in case of multiple modules
    int64_t scaleChannelOffset = 0;

    T const* scale = nullptr;

    // this loop searches for the module the current column is a part of, in case of multiple modules
    for (auto moduleId = 0; moduleId < numModules; moduleId++)
    {
        if (channelId < cumModuleSizes[moduleId])
        {
            // pick the proper scale for the token and module
            scale = scales[numTokens * moduleId + tokenId];
            break;
        }
        // adjust scale offset
        scaleChannelOffset = cumModuleSizes[moduleId];
    }

    if (threadId < numChannels * numTokens)
    {
        // apply scaling if scale is not null (it is null in case of a non-DoRA adapter)
        result[threadId] = scale == nullptr ? a[threadId] : a[threadId] * scale[channelId - scaleChannelOffset];
    }
}

template <typename T>
void tokenPerChannelScale(int64_t const numel, size_t const numModules, size_t const numTokens,
    int64_t const* __restrict__ cumModuleSizes, T const* __restrict__ a, T const* const* __restrict__ scale_ptrs,
    T* __restrict__ result, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((numel + 255) / 256);

    tokenPerChannelScaleKernel<T>
        <<<grid, block, 0, stream>>>(numModules, numTokens, cumModuleSizes, a, scale_ptrs, result);
}

template void tokenPerChannelScale<half>(int64_t const numel, size_t const numModules, size_t const numTokens,
    int64_t const* __restrict__ cumModuleSizes, half const* __restrict__ a, half const* const* __restrict__ scale_ptrs,
    half* __restrict__ result, hipStream_t stream);

#ifdef ENABLE_BF16
template void tokenPerChannelScale<hip_bfloat16>(int64_t const numel, size_t const numModules, size_t const numTokens,
    int64_t const* __restrict__ cumModuleSizes, hip_bfloat16 const* __restrict__ a,
    hip_bfloat16 const* const* __restrict__ scale_ptrs, hip_bfloat16* __restrict__ result, hipStream_t stream);
#endif

} // namespace tensorrt_llm::kernels
