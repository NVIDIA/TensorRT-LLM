#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/quantTypeUtils.cuh"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/layernormKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename Tf, typename T>
__inline__ __device__ Tf compute_layernorm(Tf val, float s_mean, float s_variance, T const* gamma, T const* beta, int i)
{
    Tf ret = (val - s_mean) * s_variance * cuda_cast<Tf>(gamma[i]);
    if (beta != nullptr)
    {
        ret = ret + cuda_cast<Tf>(beta[i]);
    }
    return ret;
}

/* Computes the layernorm https://pytorch.org/docs/stable/generated/torch.nn.LayerNorm.html
 * normed_output <- ( (input - E[input]) / Sqrt(Var[input] + eps) ) * gamma + beta
 * input is [tokens, hidden_dim]. Mean and Variance are per-row (i.e. per-token)
 *
 * One CTA handles one row.
 *
 * with USE_DIFF_OF_SQUARES set to false:
 * First pass (loop) computes the mean.
 * Second computes the variance via Var[x] = E[(x - E[x])²].
 * Third pass computes and writes normed_output
 *
 * with USE_DIFF_OF_SQUARES set to true (may be faster but less accurate):
 * First pass (loop) computes the mean and variance via Var[x] = E[x²] - E[x]²
 * Second pass computes and writes normed_output
 *
 * QuantT is the quantized data type (e.g. int8_t, __hip_fp8_e4m3_fnuz)
 *
 * USE_SHMEM controls if we cache input values into shared memory
 *
 * Optional: with dynamic scaling, the last pass doesn't write immediately but finds the
 *           amax per row. A final pass scales to int8 accordingly, and writes output to
 *           normed_output_quant.
 */
template <typename T, typename QuantT, bool USE_SHMEM, bool USE_DIFF_OF_SQUARES = false>
__global__ void generalLayerNorm(T const* input, T const* gamma, T const* beta, T* normed_output, float const eps,
    int tokens, int hidden_dim, float const* clamp_ptr, float const* scale_orig_quant_per_tensor,
    float* scale_orig_quant_per_token, float* sum_per_token, QuantT* normed_output_quant, bool has_fp8_min_scaling)
{
    constexpr auto num_elems_T = num_elems<T>::value;
    using QuantT_packed_t = typename packed_as<QuantT, num_elems_T>::type;
    using float_packed_t = typename packed_as<float, num_elems_T>::type;
    using T_scalar = typename packed_as<T, 1>::type;

    // The clamping minimum / maximum values.
    T const clamp_min = cuda_cast<T>(clamp_ptr ? clamp_ptr[0] : -FLT_MAX);
    T const clamp_max = cuda_cast<T>(clamp_ptr ? clamp_ptr[1] : FLT_MAX);

    // The quantized data type's maximum value (upper-bound).
    static constexpr float MAX_QUANT_VAL = QuantTypeStaticVals<QuantT>::MAX_VAL;
    // The minimum scaling factor (lower-bound)
    static constexpr float MIN_SCALING_FACTOR = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR;
    static constexpr float MIN_SCALING_FACTOR_RCP = QuantTypeStaticVals<QuantT>::MIN_SCALING_FACTOR_RCP;

    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T* shmem = reinterpret_cast<T*>(_shmem);
    __shared__ float s_mean;
    __shared__ float s_variance;

    int const tidx = threadIdx.x;
    int const bidx = blockIdx.x;

    float mean = 0.0f;
    float variance = 0.0f;
    float local_sum = 0.0f;
    float local_var_sum = 0.0f;

    int const n_elems = hidden_dim / num_elems_T;
    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        const T val = input[bidx * n_elems + i];
        if constexpr (USE_SHMEM)
        {
            shmem[i] = val;
        }

        const float_packed_t val_f = cuda_cast<float_packed_t>(val);
        local_sum += cuda_sum<float>(val_f);
        if constexpr (USE_DIFF_OF_SQUARES)
        {
            local_var_sum += cuda_sum<float>(val_f * val_f);
        }
    }

    if constexpr (USE_DIFF_OF_SQUARES)
    {
        float packed[2] = {local_sum, local_var_sum};
        blockReduceSumV2<float, 2>(packed);
        mean = packed[0];
        variance = packed[1];
    }
    else
    {
        mean = blockReduceSum(local_sum);
    }

    if (threadIdx.x == 0)
    {
        mean = mean / hidden_dim;
        s_mean = mean;
        if constexpr (USE_DIFF_OF_SQUARES)
        {
            variance = (variance / hidden_dim) - (mean * mean); // Var[x] = E[x²] - E[x]²
            s_variance = rsqrtf(variance + eps);
        }
    }
    __syncthreads();

    if constexpr (!USE_DIFF_OF_SQUARES)
    {
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            const T val = USE_SHMEM ? shmem[i] : input[bidx * n_elems + i];
            float_packed_t diff = cuda_cast<float_packed_t>(val) - s_mean;
            local_var_sum += cuda_sum<float>(diff * diff);
        }
        variance = blockReduceSum(local_var_sum);

        if (threadIdx.x == 0)
        {
            s_variance = rsqrtf(variance / hidden_dim + eps);
        }
        __syncthreads();
    }

    bool const with_per_token_scaling = scale_orig_quant_per_token != nullptr;
    bool const with_per_tensor_scaling = scale_orig_quant_per_tensor != nullptr;
    bool const with_per_token_sum = sum_per_token != nullptr;

    const float_packed_t scale_orig_quant
        = cuda_cast<float_packed_t>(with_per_tensor_scaling ? *scale_orig_quant_per_tensor : 0.0f);
    T_scalar amax = 1e-6f;
    local_sum = 0.f;

    for (int i = tidx; i < n_elems; i += blockDim.x)
    {
        int const index = bidx * n_elems + i;
        const float_packed_t val_f = cuda_cast<float_packed_t>(USE_SHMEM ? shmem[i] : input[index]);
        T val = cuda_cast<T>(compute_layernorm(val_f, s_mean, s_variance, gamma, beta, i));

        if (with_per_token_scaling)
        {
            val = cuda_clamp(val, clamp_min, clamp_max);
            amax = cuda_max(cuda_max<T_scalar, T>(cuda_abs(val)), amax);
            if constexpr (USE_SHMEM)
            {
                shmem[i] = val;
            }
        }
        else if (with_per_tensor_scaling)
        {
            val = cuda_clamp(val, clamp_min, clamp_max);
            reinterpret_cast<QuantT_packed_t*>(normed_output_quant)[index]
                = cuda_cast<QuantT_packed_t>(cuda_cast<float_packed_t>(val) * scale_orig_quant);
        }
        else
        {
            normed_output[index] = val;
        }

        if (with_per_token_sum)
        {
            local_sum += cuda_sum<float>(cuda_cast<float_packed_t>(val));
        }
    }

    if (with_per_token_scaling)
    {
        float abs_max_f = blockAllReduceMax(cuda_cast<float>(amax));
        float const dynamic_per_token_scale = has_fp8_min_scaling
            ? fminf(MAX_QUANT_VAL / abs_max_f, MIN_SCALING_FACTOR_RCP)
            : (MAX_QUANT_VAL / abs_max_f);
        for (int i = tidx; i < n_elems; i += blockDim.x)
        {
            int const index = bidx * n_elems + i;
            float_packed_t val_f = cuda_cast<float_packed_t>(USE_SHMEM ? shmem[i] : input[index]);
            if constexpr (!USE_SHMEM)
            {
                val_f = compute_layernorm(val_f, s_mean, s_variance, gamma, beta, i);
            }

            reinterpret_cast<QuantT_packed_t*>(normed_output_quant)[index]
                = cuda_cast<QuantT_packed_t>(val_f * cuda_cast<float_packed_t>(dynamic_per_token_scale));
        }
        if (tidx == 0)
        {
            scale_orig_quant_per_token[bidx] = has_fp8_min_scaling
                ? cuda_max(abs_max_f / MAX_QUANT_VAL, MIN_SCALING_FACTOR)
                : abs_max_f / MAX_QUANT_VAL;
        }
    }

    if (with_per_token_sum)
    {
        float packed_sum[1] = {local_sum};
        blockReduceSumV2<float, 1>(packed_sum);
        if (tidx == 0)
        {
            sum_per_token[bidx] = packed_sum[0];
        }
    }
}

template <bool USE_DIFF_OF_SQUARES, typename T, typename QuantT>
void dispatch_layernorm_type_square_method(T const* input, T const* gamma, T const* beta, T* normed_output,
    float const eps, int tokens, int hidden_dim, float const* clamp_ptr, float const* scale_orig_quant_per_tensor,
    float* scale_orig_quant_per_token, float* sum_per_token, QuantT* normed_output_quant,
    bool const has_fp8_min_scaling, dim3 const grid, dim3 const block, size_t const shmem_size, hipStream_t stream)
{
    // Do we use shared memory to cache intermediate results
    bool use_shmem = true;
    if (shmem_size >= (48 << 10))
    {
        hipError_t ret = hipFuncSetAttribute(reinterpret_cast<const void*>(generalLayerNorm<T), QuantT, true, USE_DIFF_OF_SQUARES>,
            hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size);
        // Use shared memory when the capacity is enough
        use_shmem = (ret == hipSuccess);
    }

    if (use_shmem)
    {
        generalLayerNorm<T, QuantT, true, USE_DIFF_OF_SQUARES><<<grid, block, shmem_size, stream>>>(input, gamma, beta,
            normed_output, eps, tokens, hidden_dim, clamp_ptr, scale_orig_quant_per_tensor, scale_orig_quant_per_token,
            sum_per_token, normed_output_quant, has_fp8_min_scaling);
    }
    else
    {
        generalLayerNorm<T, QuantT, false, USE_DIFF_OF_SQUARES><<<grid, block, 0, stream>>>(input, gamma, beta,
            normed_output, eps, tokens, hidden_dim, clamp_ptr, scale_orig_quant_per_tensor, scale_orig_quant_per_token,
            sum_per_token, normed_output_quant, has_fp8_min_scaling);
    }
}

template <typename T, typename QuantT>
void dispatch_layernorm_type(T const* input, T const* gamma, T const* beta, T* normed_output, float const eps,
    int tokens, int hidden_dim, float const* clamp_ptr, float const* scale_orig_quant_per_tensor,
    float* scale_orig_quant_per_token, float* sum_per_token, QuantT* normed_output_quant,
    bool const has_fp8_min_scaling, dim3 const grid, dim3 const block, size_t const shmem_size, hipStream_t stream,
    bool const use_diff_of_squares)
{
    if (use_diff_of_squares)
    {
        dispatch_layernorm_type_square_method<true>(input, gamma, beta, normed_output, eps, tokens, hidden_dim,
            clamp_ptr, scale_orig_quant_per_tensor, scale_orig_quant_per_token, sum_per_token, normed_output_quant,
            has_fp8_min_scaling, grid, block, shmem_size, stream);
    }
    else
    {
        dispatch_layernorm_type_square_method<false>(input, gamma, beta, normed_output, eps, tokens, hidden_dim,
            clamp_ptr, scale_orig_quant_per_tensor, scale_orig_quant_per_token, sum_per_token, normed_output_quant,
            has_fp8_min_scaling, grid, block, shmem_size, stream);
    }
}

template <typename T, typename QuantT>
void invokeGeneralLayerNorm(T* out, T const* input, T const* gamma, T const* beta, float const eps, int const tokens,
    int const hidden_dim, QuantMode quantMode, hipStream_t stream, bool use_diff_of_squares, float const* clamp_ptr,
    float const* scale, float* dynamic_scale, float* sum_per_token, QuantT* normed_output_quant)
{
    dim3 grid(tokens);
    dim3 block(min(hidden_dim, 1024));
    // Make sure block.x is multiple of 32 for warp shuffle to work
    block.x = 32 * ((block.x + 31) / 32);

    constexpr size_t vec_size = 2;
    const size_t shmem_size = hidden_dim * sizeof(T);
    bool const use_vec_type = (hidden_dim % vec_size == 0)
        && (std::is_same<T, half>::value
#ifdef ENABLE_BF16
            || std::is_same<T, __hip_bfloat16>::value
#endif
        );

    // Enable min_scaling factor if it is fp8 row-wise per-token quantization
    bool has_fp8_min_scaling = quantMode.hasFp8RowWise();

    if (use_vec_type)
    {
        using Tp = typename packed_as<T, vec_size>::type;
        dispatch_layernorm_type(reinterpret_cast<Tp const*>(input), reinterpret_cast<Tp const*>(gamma),
            reinterpret_cast<Tp const*>(beta), reinterpret_cast<Tp*>(out), eps, tokens, hidden_dim, clamp_ptr, scale,
            dynamic_scale, sum_per_token, normed_output_quant, has_fp8_min_scaling, grid, block, shmem_size, stream,
            use_diff_of_squares);
    }
    else
    {
        dispatch_layernorm_type(input, gamma, beta, out, eps, tokens, hidden_dim, clamp_ptr, scale, dynamic_scale,
            sum_per_token, normed_output_quant, has_fp8_min_scaling, grid, block, shmem_size, stream,
            use_diff_of_squares);
    }
}

#define INSTANTIATE_GENERAL_LAYERNORM(T, QuantT)                                                                       \
    template void invokeGeneralLayerNorm(T* out, const T* input, const T* gamma, const T* beta, const float eps,       \
        const int tokens, const int hidden_dim, QuantMode quantMode, hipStream_t stream, bool use_diff_of_squares,    \
        const float* clamp_ptr, float const* scale, float* dynamic_scale, float* sum_per_token,                        \
        QuantT* normed_output_quant);

INSTANTIATE_GENERAL_LAYERNORM(float, int8_t);
INSTANTIATE_GENERAL_LAYERNORM(half, int8_t);

#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_LAYERNORM(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_GENERAL_LAYERNORM(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_GENERAL_LAYERNORM(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_GENERAL_LAYERNORM(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

} // namespace kernels
} // namespace tensorrt_llm
