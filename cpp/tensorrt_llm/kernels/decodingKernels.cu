#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/decodingKernels.h"
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm
{

namespace kernels
{

class CopyBeamHypothesesStruct
{
public:
    TokenIdType const* srcOutputIdsCBA; // [BS, BM*2, MSL]
    TokenIdType* dstOutputIdsCBA;       // [BS, BM*2, MSL]
    SizeType32 outputIdsNumElts;

    float const* srcLogProbsCBA; // [BS, BM*2, MSL]
    float* dstLogProbsCBA;       // [BS, BM*2, MSL]
    SizeType32 logProbsNumElts;

    SizeType32 const* srcSequenceLengthsCBA; // [BS, BM*2]
    SizeType32* dstSequenceLengthsCBA;       // [BS, BM*2]
    SizeType32 sequenceLengthsNumElts;

    float const* srcCumLogProbsCBA; // [BS, BM*2]
    float* dstCumLogProbsCBA;       // [BS, BM*2]
    SizeType32 cumLogProbsCBANumElts;

    float const* srcNormedScoresCBA; // [BS, BM*2]
    float* dstNormedScoresCBA;       // [BS, BM*2]
    SizeType32 normedScoresNumElts;

    SizeType32 const* srcNumBeamsCBA; // [BS]
    SizeType32* dstNumBeamsCBA;       // [BS]
    SizeType32 numBeamsNumElts;

    float const* srcMinNormedScoresCBA; // [BS]
    float* dstMinNormedScoresCBA;       // [BS]
    SizeType32 minNormedScoresNumElts;

    bool const* srcBatchDones; // [BS]
    bool* dstBatchDones;       // [BS]
    SizeType32 batchDonesNumElts;

    float const* srcCumLogProbs; // [BS, BM]
    float* dstCumLogProbs;       // [BS, BM]
    SizeType32 cumLogProbsNumElts;
};

__global__ void gatherTree(gatherTreeParam param)
{
    for (int batchbeamIdx = blockIdx.x * blockDim.x + threadIdx.x; batchbeamIdx < param.batchSize * param.beamWidth;
         batchbeamIdx += gridDim.x * blockDim.x)
    {
        int const batch = batchbeamIdx / param.beamWidth;
        int const beam = batchbeamIdx % param.beamWidth;
        int const inputLen = param.inputLengths == nullptr ? 0 : param.inputLengths[batchbeamIdx];

        int const* parentIds = param.parentIds;
        int const* stepIds = param.stepIds;

        // TODO optimize the reduce_max operation for large beamWidth
        int maxLen = -1;
        bool updateResponseInputLength = param.responseInputLengths != nullptr;
        // int selected_beam_index = 0;
        for (int beamIdx = 0; beamIdx < param.beamWidth; beamIdx++)
        {
            int tmpLen
                = param.sequenceLengths[batch * param.beamWidth + beamIdx] + param.maxSequenceLengthFinalStep - 1;
            param.sequenceLengths[batch * param.beamWidth + beamIdx] = tmpLen;
            if (updateResponseInputLength)
            {
                param.responseInputLengths[batch * param.beamWidth + beamIdx] = inputLen;
            }
            if (tmpLen > maxLen)
            {
                maxLen = tmpLen;
            }
        }
        int const maxSeqLenB = min(param.maxSeqLen, maxLen);
        if (maxSeqLenB <= 0)
        {
            continue;
        }

        int const initialTgtIx = batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + maxSeqLenB - 1;
        int const initialParentIx = batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + maxSeqLenB - 1;
        param.outputIds[initialTgtIx] = __ldg(stepIds + initialParentIx);
        int parent = parentIds == nullptr ? 0 : __ldg(parentIds + initialParentIx) % param.beamWidth;
        bool foundBad = false;

        for (int level = maxSeqLenB - 2; level >= 0; --level)
        {
            int const levelBeamIx = batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + level;
            int const levelParentIx = batch * param.beamWidth * param.maxSeqLen + parent * param.maxSeqLen + level;
            if (parent < 0 || parent > param.beamWidth)
            {
                param.outputIds[levelBeamIx] = param.endTokens[batch];
                parent = -1;
                foundBad = true;
            }
            else
            {
                param.outputIds[levelBeamIx] = __ldg(stepIds + levelParentIx);
                parent = parentIds == nullptr ? 0 : __ldg(parentIds + levelParentIx) % param.beamWidth;
            }
        }
        // set the padded part as end_token
        // inputLen
        for (int index = maxLen; index < param.maxSeqLen; ++index)
        {
            param.outputIds[batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + index]
                = param.endTokens[batch];
        }

        // Not necessary when using a BeamSearchDecoder, but necessary
        // when a user feeds in possibly broken trajectory (i.e., non-eos
        // entries in a beam following eos entries).
        if (!foundBad)
        {
            bool finished = false;
            // skip the step 0 because it is often the start token
            int startStep = 1;
            for (int time = startStep; time < maxSeqLenB; ++time)
            {
                int const levelBeamIx = batch * param.beamWidth * param.maxSeqLen + beam * param.maxSeqLen + time;
                if (finished)
                {
                    param.outputIds[levelBeamIx] = param.endTokens[batch];
                }
                else if (param.outputIds[levelBeamIx] == param.endTokens[batch])
                {
                    finished = true;
                }
            }
        }
    }
}

struct RankNorm
{
    int rank;
    float norm;
};

inline __device__ RankNorm swap(RankNorm const& rankNorm, int mask, int dir)
{
    // Exchange RankNorm data inside the warp
    RankNorm other;
    other.rank = __shfl_xor_sync(unsigned(-1), rankNorm.rank, mask);
    other.norm = __shfl_xor_sync(unsigned(-1), rankNorm.norm, mask);
    // dir == 0 -> return larger one
    // dir == 1 -> return smaller one
    bool doSwap = (rankNorm.norm != other.norm) && ((rankNorm.norm > other.norm) == dir);
    return doSwap ? other : rankNorm;
}

inline __device__ uint32_t bfe(uint32_t a, uint32_t start, uint32_t len = 1)
{
    uint32_t d;
    asm volatile("bfe.u32 %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(start), "r"(len));
    return d;
}

__global__ void finalized(gatherTreeParam param)
{
    int const beamIdx = static_cast<int>(threadIdx.x);
    int const beamWidth{param.beamWidth};

    extern __shared__ char array[];
    int* sRank = (int*) (array);
    int* sLength = (int*) (sRank + beamWidth);
    float* sScores = (float*) (sLength + beamWidth);
    float* sNormedScores = (float*) (sScores + beamWidth);
    int* sIds = (int*) (sNormedScores + beamWidth);

    if (beamIdx < beamWidth)
    {
        int const idx = blockIdx.x * param.beamWidth + beamIdx;
        int const numGeneratedToken{param.sequenceLengths[idx] - param.inputLengths[idx]};
        sNormedScores[beamIdx] = applyLengthPenalty(param.cumLogProbs[idx], numGeneratedToken, param.lengthPenalty);
        sLength[beamIdx] = param.sequenceLengths[idx];
        sScores[beamIdx] = param.cumLogProbs[idx];
    }
    for (int idx = beamIdx; idx < beamWidth * param.maxSeqLen; idx += blockDim.x)
    {
        sIds[idx] = param.outputIds[blockIdx.x * param.beamWidth * param.maxSeqLen + idx];
    }
    __syncthreads();

    RankNorm rankNorm;
    rankNorm.rank = beamIdx;
    rankNorm.norm = beamIdx < beamWidth ? sNormedScores[beamIdx] : -FLT_MAX;

    if (beamWidth < 32)
    {
        int warpid = threadIdx.x / 32;
        int laneid = threadIdx.x % 32;

        if (warpid == 0 && beamWidth > 1)
        {
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 1) ^ bfe(laneid, 0)); //  2
        }

        if (warpid == 0 && beamWidth > 2)
        {
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 2) ^ bfe(laneid, 1)); //  3~4
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 2) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && beamWidth > 4)
        {
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 3) ^ bfe(laneid, 2)); //  5~8
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 3) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 3) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && beamWidth > 8)
        {
            rankNorm = swap(rankNorm, 0x08, bfe(laneid, 4) ^ bfe(laneid, 3)); // 9~16
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 4) ^ bfe(laneid, 2));
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 4) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 4) ^ bfe(laneid, 0));
        }

        if (warpid == 0 && beamWidth > 16)
        {
            rankNorm = swap(rankNorm, 0x10, bfe(laneid, 5) ^ bfe(laneid, 4)); // 17~32
            rankNorm = swap(rankNorm, 0x08, bfe(laneid, 5) ^ bfe(laneid, 3));
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 5) ^ bfe(laneid, 2));
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 5) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 5) ^ bfe(laneid, 0));
        }
    }
    else
    {
        // Not supported! We must have a check before calling that kernel.
    }

    if (beamIdx < beamWidth)
    {
        sRank[beamIdx] = rankNorm.rank;
    }

    __syncthreads();

    if (beamIdx < beamWidth)
    {
        auto srcIdx{rankNorm.rank};
        auto tgtIdx{blockIdx.x * param.beamWidth + beamIdx};
        param.sequenceLengths[tgtIdx] = sLength[srcIdx];
        param.cumLogProbs[tgtIdx] = sScores[srcIdx];
    }

    for (int beamIdx = 0; beamIdx < beamWidth; beamIdx++)
    {
        for (int i = threadIdx.x; i < sLength[sRank[beamIdx]]; i += blockDim.x)
        {
            param.outputIds[blockIdx.x * beamWidth * param.maxSeqLen + beamIdx * param.maxSeqLen + i]
                = sIds[sRank[beamIdx] * param.maxSeqLen + i];
        }
    }
}

void invokeGatherTree(gatherTreeParam param)
{
    int batchbeam = param.batchSize * param.beamWidth;
    dim3 grid(1), block(batchbeam);
    // though decoder do not support > 1024 for now
    if (batchbeam > 1024)
    {
        grid.x = ceil(param.batchSize * param.beamWidth / 1024.);
        block.x = 1024;
    }
    gatherTree<<<grid, block, 0, param.stream>>>(param);
    sync_check_cuda_error(param.stream);

    if (param.beamWidth > 1)
    {
        TLLM_CHECK_WITH_INFO(param.beamWidth <= 32, "TRT-LLM does not support beam width > 32 now");
        // sort results by normalized cumLogProbs
        dim3 grid(param.batchSize);
        dim3 block(divUp(param.beamWidth, 32) * 32);

        auto shm_size = param.beamWidth * (sizeof(float) * 2 + sizeof(int) * 2 + sizeof(int) * param.maxSeqLen);
        finalized<<<grid, block, shm_size, param.stream>>>(param);
    }
}

__global__ void insertUnfinishedPathKernel(BeamHypotheses bh)
{
    // Move ALL unfinished beams from bh.outputIdsUnfinish to bh.outputIdsCBA
    // So here might be more than `nBM` beams in bh.outputIdsCBA after this kernel
    // Data movement:
    // bh.outputIdsUnfinish -> bh.outputIdsCBA
    // bh.sequenceLengths   -> bh.sequenceLengthsCBA
    // bh.cumLogProbs       -> bh.cumLogProbsCBA
    // bh.logProbsTiled     -> bh.logProbsCBA
    // update bh.normedScoresCBA
    // update bh.numBeamsCBA

    size_t const bid = blockIdx.x;       // Index of Batch
    size_t const nBM{bh.nBeamWidth};
    size_t const nMBS{bh.nMaxBatchSize}; // Only for bh.logProbsTiled
    size_t const nMSL{bh.nMaxSeqLen};
    bool const bOutputLogProbs{bh.logProbsCBA != nullptr && bh.logProbsTiled != nullptr};
    int const indexDstStart{bh.numBeamsCBA[bid]};

    if (bh.batchDones[bid])
    {
        return;
    }

    for (int i = 0; i < nBM; ++i)
    {
        int const srcBeam = bid * nBM + i;
        int const dstBeam = bid * nBM * 2 + i + indexDstStart;
        int const step = bh.sequenceLengths[srcBeam] - 1;

        // The last token
        int const srcId = srcBeam * nMSL + step;
        int const dstId = dstBeam * nMSL + step;
        bh.outputIdsCBA[dstId] = bh.outputIdsUnfinish[srcId];
        if (bOutputLogProbs)
        {
            bh.logProbsCBA[dstId] = bh.logProbsTiled[step * nMBS * nBM + srcBeam];
        }
        // Previous tokens
        int prevId = bh.parentIdsUnfinish[srcId];
        for (int j = step - 1; j >= 0; --j)
        {
            int const index = bid * nBM * nMSL + prevId * nMSL + j;
            bh.outputIdsCBA[dstBeam * nMSL + j] = bh.outputIdsUnfinish[index];
            prevId = bh.parentIdsUnfinish[index];
        }
        if (bOutputLogProbs)
        {
            prevId = bh.parentIdsUnfinish[srcId];
            for (int j = step - 1; j >= 0; --j)
            {
                int const index = bid * nBM * nMSL + prevId * nMSL + j;
                bh.logProbsCBA[dstBeam * nMSL + j] = bh.logProbsTiled[j * nMBS * nBM + bid * nBM + prevId];
                prevId = bh.parentIdsUnfinish[index];
            }
        }
        // Other parameters
        bh.sequenceLengthsCBA[dstBeam] = bh.sequenceLengths[srcBeam];
        bh.normedScoresCBA[dstBeam]
            = applyLengthPenalty(bh.cumLogProbs[srcBeam], step - bh.inputLengths[srcBeam] + 1, bh.lengthPenalties[bid]);
        bh.cumLogProbsCBA[dstBeam] = bh.cumLogProbs[srcBeam];
        bh.numBeamsCBA[bid]++;
    }
}

void invokeInsertUnfinishedPath(BeamHypotheses& bh, hipStream_t stream)
{
    insertUnfinishedPathKernel<<<bh.nBatchSize, 1, 0, stream>>>(bh);
}

__global__ void finalizeKernel(BeamHypotheses bh)
{
    // Do index sort on bh.normedScoresCBA, then move buffers from CBA to output by the order of index
    // Data movement:
    // bh.outputIdsCBA       -> bh.outputIds
    // bh.sequenceLengthsCBA -> bh.sequenceLengths
    // bh.cumLogProbsCBA     -> bh.cumLogProbs
    // bh.logProbsCBA        -> bh.logProbs

    int const bid = blockIdx.x;  // Index of Batch
    int const tid = threadIdx.x; // Index of Beam
    size_t const nBM{bh.nBeamWidth};
    size_t const nMSL{bh.nMaxSeqLen};
    int const nCBA{bh.numBeamsCBA[bid]}; // Count of candidates in CBA, nBM <= nCBA <= 2*nBM

    extern __shared__ char smem[];
    int* smemRank = (int*) (smem);                // [nBM]
    float* smemScore = (float*) (smemRank + nBM); // [2*nBM]
    int* smemSL = (int*) (smemScore + nBM * 2);   // [nBM]

    // Sort
    for (int i = tid; i < nCBA; i += blockDim.x)
    {
        smemScore[i] = bh.normedScoresCBA[bid * nBM * 2 + i];
    }
    __syncthreads();

    if (nCBA <= 32)
    {
        int const warpid = tid / 32;
        int const laneid = tid % 32;
        RankNorm rankNorm{tid, tid < nCBA ? smemScore[tid] : -FLT_MAX};

        if (warpid == 0 && nCBA > 1)
        {
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 1) ^ bfe(laneid, 0)); // 2
        }
        if (warpid == 0 && nCBA > 2)
        {
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 2) ^ bfe(laneid, 1)); // 3~4
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 2) ^ bfe(laneid, 0));
        }
        if (warpid == 0 && nCBA > 4)
        {
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 3) ^ bfe(laneid, 2)); // 5~8
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 3) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 3) ^ bfe(laneid, 0));
        }
        if (warpid == 0 && nCBA > 8)
        {
            rankNorm = swap(rankNorm, 0x08, bfe(laneid, 4) ^ bfe(laneid, 3)); // 9~16
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 4) ^ bfe(laneid, 2));
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 4) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 4) ^ bfe(laneid, 0));
        }
        if (warpid == 0 && nCBA > 16)
        {
            rankNorm = swap(rankNorm, 0x10, bfe(laneid, 5) ^ bfe(laneid, 4)); // 17~32
            rankNorm = swap(rankNorm, 0x08, bfe(laneid, 5) ^ bfe(laneid, 3));
            rankNorm = swap(rankNorm, 0x04, bfe(laneid, 5) ^ bfe(laneid, 2));
            rankNorm = swap(rankNorm, 0x02, bfe(laneid, 5) ^ bfe(laneid, 1));
            rankNorm = swap(rankNorm, 0x01, bfe(laneid, 5) ^ bfe(laneid, 0));
        }
        if (tid < nBM)
        {
            smemRank[tid] = rankNorm.rank;
        }
        __syncthreads();
    }
    else
    {
        // TODO, wili: use CUB to sort for large nCBA
        for (int i = 0; i < nBM; ++i)
        {
            float maxScore = -FLT_MAX;
            for (int j = 0; j < (nCBA + 1024 - 1) / 1024; ++j)
            {
                int const index = tid + 1024 * j;
                float const score = (index < bh.numBeamsCBA[bid]) ? smemScore[index] : -FLT_MAX;
                float const maxScore1 = blockReduceMax<float>(score);
                maxScore = max(maxScore, maxScore1);
            }
            if (tid == 0)
            {
                for (int j = 0; j < nCBA; ++j)
                {
                    if (smemScore[j] == maxScore)
                    {
                        smemRank[i] = j;
                        smemScore[j] = -FLT_MAX;
                        break;
                    }
                }
            }
            __syncthreads();
        }
    }

    // Move bh.sequenceLengths, bh.cumLogProbs
    if (tid < nBM)
    {
        smemSL[tid] = bh.sequenceLengthsCBA[bid * nBM * 2 + smemRank[tid]];
        bh.sequenceLengths[bid * nBM + tid] = smemSL[tid];
        if (bh.cumLogProbs != nullptr)
        {
            bh.cumLogProbs[bid * nBM + tid] = bh.cumLogProbsCBA[bid * nBM * 2 + smemRank[tid]];
        }
    }
    __syncthreads();

    // Move bh.outputIds, bh.logProbs
    for (int beamIdx = 0; beamIdx < nBM; beamIdx++)
    {
        for (int i = tid; i < smemSL[beamIdx]; i += blockDim.x)
        {
            int const dst = bid * nBM * nMSL + beamIdx * nMSL + i;
            int const src = bid * nBM * 2 * nMSL + smemRank[beamIdx] * nMSL + i;
            bh.outputIds[dst] = bh.outputIdsCBA[src];
        }
        if (bh.logProbs != nullptr)
        {
            for (int i = tid; i < smemSL[beamIdx]; i += blockDim.x)
            {
                if (int const inputLength = bh.inputLengths[bid * nBM + beamIdx]; i >= inputLength)
                {
                    int const dst = bid * nBM * nMSL + beamIdx * nMSL + i;
                    int const src = bid * nBM * 2 * nMSL + smemRank[beamIdx] * nMSL + i;
                    bh.logProbs[dst - inputLength] = bh.logProbsCBA[src];
                }
            }
        }
    }
}

void invokeFinalize(BeamHypotheses& bh, hipStream_t stream)
{
    TLLM_LOG_TRACE("%s %s start", __FILE__, __PRETTY_FUNCTION__);

    int const nBM = bh.nBeamWidth;
    int const nThread = min(roundUp(nBM * 2, 32), 1024);
    size_t const nByteSharedMemory = (sizeof(int) + sizeof(float)) * nBM * 2;
    finalizeKernel<<<bh.nBatchSize, nThread, nByteSharedMemory, stream>>>(bh);
    sync_check_cuda_error(stream);

    TLLM_LOG_TRACE("%s %s stop", __FILE__, __PRETTY_FUNCTION__);
}

__global__ void copyBeamHypotheses(CopyBeamHypothesesStruct copyStruct)
{
    auto const idx = static_cast<SizeType32>(threadIdx.x + blockIdx.x * blockDim.x);
    auto const stride = static_cast<SizeType32>(blockDim.x * gridDim.x);

    for (SizeType32 ii = idx; ii < copyStruct.outputIdsNumElts; ii += stride)
    {
        copyStruct.dstOutputIdsCBA[ii] = copyStruct.srcOutputIdsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.logProbsNumElts; ii += stride)
    {
        copyStruct.dstLogProbsCBA[ii] = copyStruct.srcLogProbsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.cumLogProbsNumElts; ii += stride)
    {
        copyStruct.dstCumLogProbs[ii] = copyStruct.srcCumLogProbs[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.sequenceLengthsNumElts; ii += stride)
    {
        copyStruct.dstSequenceLengthsCBA[ii] = copyStruct.srcSequenceLengthsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.cumLogProbsCBANumElts; ii += stride)
    {
        copyStruct.dstCumLogProbsCBA[ii] = copyStruct.srcCumLogProbsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.normedScoresNumElts; ii += stride)
    {
        copyStruct.dstNormedScoresCBA[ii] = copyStruct.srcNormedScoresCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.numBeamsNumElts; ii += stride)
    {
        copyStruct.dstNumBeamsCBA[ii] = copyStruct.srcNumBeamsCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.minNormedScoresNumElts; ii += stride)
    {
        copyStruct.dstMinNormedScoresCBA[ii] = copyStruct.srcMinNormedScoresCBA[ii];
    }

    for (SizeType32 ii = idx; ii < copyStruct.batchDonesNumElts; ii += stride)
    {
        copyStruct.dstBatchDones[ii] = copyStruct.srcBatchDones[ii];
    }
}

void invokeCopyBeamHypotheses(DecodingOutput::BeamHypotheses const& src, DecodingOutput::BeamHypotheses const& dst,
    ITensor& srcCumLogProbs, ITensor& dstCumLogProbs, runtime::CudaStream const& stream, SizeType32 numSMs)
{
    CopyBeamHypothesesStruct copyStruct = {};

    copyStruct.srcOutputIdsCBA = bufferCast<TokenIdType>(*(src.outputIdsCBA));
    copyStruct.dstOutputIdsCBA = bufferCast<TokenIdType>(*(dst.outputIdsCBA));
    copyStruct.outputIdsNumElts = dst.outputIdsCBA->getSize();

    copyStruct.srcLogProbsCBA = bufferCast<float>(*(src.logProbsCBA));
    copyStruct.dstLogProbsCBA = bufferCast<float>(*(dst.logProbsCBA));
    copyStruct.logProbsNumElts = dst.logProbsCBA->getSize();

    copyStruct.srcSequenceLengthsCBA = bufferCast<SizeType32>(*(src.sequenceLengthsCBA));
    copyStruct.dstSequenceLengthsCBA = bufferCast<SizeType32>(*(dst.sequenceLengthsCBA));
    copyStruct.sequenceLengthsNumElts = dst.sequenceLengthsCBA->getSize();

    copyStruct.srcCumLogProbsCBA = bufferCast<float>(*(src.cumLogProbsCBA));
    copyStruct.dstCumLogProbsCBA = bufferCast<float>(*(dst.cumLogProbsCBA));
    copyStruct.cumLogProbsCBANumElts = dst.cumLogProbsCBA->getSize();

    copyStruct.srcNormedScoresCBA = bufferCast<float>(*(src.normedScoresCBA));
    copyStruct.dstNormedScoresCBA = bufferCast<float>(*(dst.normedScoresCBA));
    copyStruct.normedScoresNumElts = dst.normedScoresCBA->getSize();

    copyStruct.srcNumBeamsCBA = bufferCast<SizeType32>(*(src.numBeamsCBA));
    copyStruct.dstNumBeamsCBA = bufferCast<SizeType32>(*(dst.numBeamsCBA));
    copyStruct.numBeamsNumElts = dst.numBeamsCBA->getSize();

    copyStruct.srcMinNormedScoresCBA = bufferCast<float>(*(src.minNormedScoresCBA));
    copyStruct.dstMinNormedScoresCBA = bufferCast<float>(*(dst.minNormedScoresCBA));
    copyStruct.minNormedScoresNumElts = dst.minNormedScoresCBA->getSize();

    copyStruct.srcBatchDones = bufferCast<bool>(*(src.batchDones));
    copyStruct.dstBatchDones = bufferCast<bool>(*(dst.batchDones));
    copyStruct.batchDonesNumElts = dst.batchDones->getSize();

    copyStruct.srcCumLogProbs = bufferCast<float>(srcCumLogProbs);
    copyStruct.dstCumLogProbs = bufferCast<float>(dstCumLogProbs);
    copyStruct.cumLogProbsNumElts = srcCumLogProbs.getSize();

    copyBeamHypotheses<<<numSMs, 256, 0, stream.get()>>>(copyStruct);
}

__global__ void initializeOutput(
    TokenIdType* finalOutputIds, TokenIdType const* endIds, SizeType32 const beam, SizeType32 const nMaxSeqLen)
{
    for (int i = threadIdx.x; i < nMaxSeqLen; i += blockDim.x)
    {
        finalOutputIds[blockIdx.x * nMaxSeqLen + i] = endIds[blockIdx.x / beam];
    }
}

void invokeInitializeOutput(TokenIdType* finalOutputIds, TokenIdType const* endIds, SizeType32 const batch,
    SizeType32 const beam, SizeType32 const nMaxSeqLen, hipStream_t stream)
{
    initializeOutput<<<batch * beam, 256, 0, stream>>>(finalOutputIds, endIds, beam, nMaxSeqLen);
}

__global__ void copyNextStepIds(TokenIdType* nextStepIds, TokenIdType const* const* outputIdsPtr,
    SizeType32 const* sequenceLengths, SizeType32 const* numNewTokens, SizeType32 const* batchSlots,
    SizeType32 batchSize, SizeType32 maxBatchSize, SizeType32 beamWidth, SizeType32 maxSeqLen,
    SizeType32 maxTokensPerStep)
{
    for (auto index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
         index < batchSize * beamWidth * maxTokensPerStep; index += static_cast<SizeType32>(blockDim.x * gridDim.x))
    {
        // numNewTokens == nullptr when Medusa is disabled
        auto const batchIdx{index / (beamWidth * maxTokensPerStep)};
        auto const batchSlot{batchSlots[batchIdx]};
        auto const remainder{index % (beamWidth * maxTokensPerStep)};
        auto const beamIdx{remainder / maxTokensPerStep};
        auto const tokenIdx{remainder % maxTokensPerStep};
        auto const newTokens{numNewTokens == nullptr ? 1 : numNewTokens[batchSlot]};
        auto const batchBeamIdx = batchSlot * beamWidth + beamIdx;
        auto const tokenBatchBeamIdx = tokenIdx * maxBatchSize * beamWidth + batchSlot * beamWidth + beamIdx;
        auto const indexSrc = sequenceLengths[batchBeamIdx] - newTokens + tokenIdx;
        if (tokenIdx >= newTokens || indexSrc < 0)
        {
            continue;
        }
        nextStepIds[tokenBatchBeamIdx] = outputIdsPtr[batchSlot][beamIdx * maxSeqLen + indexSrc];
    }
}

void invokeCopyNextStepIds(TokenIdType* nextStepIds, TokenIdType const* const* outputIdsPtr,
    SizeType32 const* sequenceLengths, SizeType32 const* numNewTokens, SizeType32 const* batchSlots,
    SizeType32 batchSize, SizeType32 maxBatchSize, SizeType32 beamWidth, SizeType32 maxSeqLen,
    SizeType32 maxTokensPerStep, hipStream_t stream)
{
    int const numElems = batchSize * beamWidth * maxTokensPerStep;
    dim3 block(min(256, numElems));
    dim3 grid(divUp(numElems, block.x));
    copyNextStepIds<<<grid, block, 0, stream>>>(nextStepIds, outputIdsPtr, sequenceLengths, numNewTokens, batchSlots,
        batchSize, maxBatchSize, beamWidth, maxSeqLen, maxTokensPerStep);
}

__global__ void transposeLogProbs(float* outputLogProbs, float* outputLogProbsTiled, SizeType32 const* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 maxBatchSize, SizeType32 beamWidth,
    SizeType32 maxSeqLen)
{
    auto index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);

    auto const batchIdx = index / (beamWidth * maxSeqLen);
    auto const tmpIdx = index % (beamWidth * maxSeqLen);
    auto const beamIdx = tmpIdx / maxSeqLen;
    auto const pos = tmpIdx % maxSeqLen;
    if (batchIdx >= batchSize)
    {
        return;
    }

    auto const batchSlot = batchSlots[batchIdx];
    if (pos < sequenceLengths[batchSlot])
    {
        auto const batchBeamIdx = batchSlot * beamWidth * maxSeqLen + beamIdx * maxSeqLen + pos;
        outputLogProbs[batchBeamIdx]
            = outputLogProbsTiled[pos * maxBatchSize * beamWidth + batchSlot * beamWidth + beamIdx];
    }
}

void invokeTransposeLogProbs(float* outputLogProbs, float* outputLogProbsTiled, SizeType32 const* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 maxBatchSize, SizeType32 beamWidth,
    SizeType32 maxSeqLen, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid(divUp(batchSize * beamWidth * maxSeqLen, block.x));
    transposeLogProbs<<<grid, block, 0, stream>>>(outputLogProbs, outputLogProbsTiled, sequenceLengths, batchSlots,
        batchSize, maxBatchSize, beamWidth, maxSeqLen);
}

} // namespace kernels

namespace runtime::kernels
{
// Must be similar to [cpp/tensorrt_llm/thop/gatherTreeOp.cpp] gatherTree
void gatherTree(DecodingOutput const& decodingOutput, DecodingInput const& decodingInput, BufferManager const& manager,
    SamplingConfig const& samplingConfig)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    auto& finalOutputIds = *decodingOutput.gatheredIds;
    auto const& finalOutputIdsShape = finalOutputIds.getShape();
    auto const& decodingOutputIdsShape = decodingOutput.ids->getShape();
    auto const batchSize = finalOutputIdsShape.d[0];
    auto const beamWidth = finalOutputIdsShape.d[1];
    auto const maxSeqLength = finalOutputIdsShape.d[2];

    TLLM_CHECK_WITH_INFO(beamWidth > 1, "gatherTree is only needed for beam search.");

    TLLM_CHECK_WITH_INFO(decodingOutputIdsShape.d[0] == batchSize,
        common::fmtstr("Decoder batch size (" FMT_DIM ") does not match final batch size (" FMT_DIM ")",
            decodingOutputIdsShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(decodingOutputIdsShape.d[1] == beamWidth,
        common::fmtstr("Decoder beam width (" FMT_DIM ") does not match final beam width (" FMT_DIM ")",
            decodingOutputIdsShape.d[1], beamWidth));
    TLLM_CHECK_WITH_INFO(decodingOutputIdsShape.d[2] <= maxSeqLength,
        common::fmtstr("Decoder seq length size (" FMT_DIM ") is too large for final seq length (" FMT_DIM ")",
            decodingOutputIdsShape.d[2], maxSeqLength));

    auto const& stream = manager.getStream().get();

    // prefill finalOutputIds with the EOS tokens from decodingInput.endIds
    tensorrt_llm::kernels::invokeInitializeOutput(bufferCast<TokenIdType>(finalOutputIds),
        bufferCast<TokenIdType>(*decodingInput.endIds), batchSize, beamWidth, maxSeqLength, stream);
    sync_check_cuda_error(stream);

    std::vector<float> lengthPenaltyVec;
    auto lengthPenaltyPtr = std::shared_ptr(manager.gpu(ITensor::makeShape({batchSize}), TRTDataType<float>::value));
    if (!samplingConfig.lengthPenalty.has_value() || samplingConfig.lengthPenalty.value().size() == 0)
    {
        lengthPenaltyVec = std::vector<float>(batchSize, 1.0f);
    }
    else if (long int const size = samplingConfig.lengthPenalty.value().size(); size == 1)
    {
        lengthPenaltyVec = std::vector<float>(batchSize, samplingConfig.lengthPenalty.value()[0]);
    }
    else
    {
        TLLM_CHECK_WITH_INFO(size == batchSize,
            common::fmtstr("Size of lengthPenalty in SamplingConfig (" FMT_DIM ") is different from batchSize (" FMT_DIM
                           ")",
                size, batchSize));
        lengthPenaltyVec = samplingConfig.lengthPenalty.value();
    }

    lengthPenaltyPtr = manager.copyFrom(lengthPenaltyVec, ITensor::makeShape({batchSize}), runtime::MemoryType::kGPU);

    tensorrt_llm::kernels::BeamHypotheses bh;
    bh.nMaxBatchSize = batchSize;
    bh.nBatchSize = batchSize;
    bh.nBeamWidth = beamWidth;
    bh.nMaxSeqLen = maxSeqLength;
    bh.lengthPenalties = bufferCast<float>(*lengthPenaltyPtr);
    bh.inputLengths = bufferCast<SizeType32>(*decodingInput.lengths);
    bh.outputIds = bufferCast<TokenIdType>(finalOutputIds);
    bh.logProbs = bufferCastOrNull<float>(decodingOutput.logProbs);
    bh.logProbsTiled = bufferCast<float>(*decodingOutput.logProbsTiled);
    bh.sequenceLengths = bufferCast<SizeType32>(*decodingOutput.lengths);
    bh.cumLogProbs = bufferCast<float>(*decodingOutput.cumLogProbs);
    bh.outputIdsCBA = bufferCast<TokenIdType>(*decodingOutput.beamHypotheses.outputIdsCBA);
    bh.logProbsCBA = bufferCast<float>(*decodingOutput.beamHypotheses.logProbsCBA);
    bh.sequenceLengthsCBA = bufferCast<SizeType32>(*decodingOutput.beamHypotheses.sequenceLengthsCBA);
    bh.cumLogProbsCBA = bufferCast<float>(*decodingOutput.beamHypotheses.cumLogProbsCBA);
    bh.normedScoresCBA = bufferCast<float>(*decodingOutput.beamHypotheses.normedScoresCBA);
    bh.numBeamsCBA = bufferCast<SizeType32>(*decodingOutput.beamHypotheses.numBeamsCBA);
    bh.minNormedScoresCBA = bufferCast<float>(*decodingOutput.beamHypotheses.minNormedScoresCBA);
    bh.batchDones = bufferCast<bool>(*decodingOutput.beamHypotheses.batchDones);
    bh.finished = bufferCast<tensorrt_llm::kernels::FinishedState>(*decodingOutput.finishReasons);
    bh.outputIdsUnfinish = bufferCast<TokenIdType>(*decodingOutput.ids);
    bh.parentIdsUnfinish = bufferCast<TokenIdType>(*decodingOutput.parentIds);

    // This is where transpose is done
    tensorrt_llm::kernels::invokeInsertUnfinishedPath(bh, stream);
    sync_check_cuda_error(stream);

    tensorrt_llm::kernels::invokeFinalize(bh, stream);
    sync_check_cuda_error(stream);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

} // namespace runtime::kernels

} // namespace tensorrt_llm
