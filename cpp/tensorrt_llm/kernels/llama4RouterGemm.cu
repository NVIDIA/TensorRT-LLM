#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/kernels/llama4RouterGemm.h"
#include "tensorrt_llm/kernels/llama4Utils.cuh"

namespace tensorrt_llm::kernels::llama4_router_gemm
{

struct __align__(8) aligned_bf16x4
{
    __align__(8) __hip_bfloat16 data[VEC_SIZE];
};

// This is the hand-optimized kernel by Po-Han.
__global__ void gemv_kernel(int num_tokens,
    __hip_bfloat16 const* __restrict__ A, // Input vector [num_tokens][5120]
    __hip_bfloat16 const* __restrict__ B, // Input matrix [128][5120]
    __hip_bfloat16* __restrict__ C        // Output vector [num_tokens][128]
)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900) && (__CUDA_ARCH__ < 1200))
    // Shared memory for block reduction
    __shared__ float reduce_buffer[BLOCK_SIZE];

    // Each thread accumulates its partial sum
    float2 thread_sum;
    thread_sum.x = 0.0f;
    thread_sum.y = 0.0f;

    // Each thread processes 4 elements at a time, 5 times
    int const token_idx = blockIdx.x / NUM_EXPERTS;
    int const row = blockIdx.x % NUM_EXPERTS; // Matrix row / Output element index
    int const tid = threadIdx.x;              // Thread ID within the block

    // FDL prefetch all B data
    aligned_bf16x4 b_vec[GEMM_K / BLOCK_SIZE / VEC_SIZE];
#pragma unroll
    for (int chunk = 0; chunk < GEMM_K / BLOCK_SIZE / VEC_SIZE; chunk++)
    {
        // Base index for this chunk
        int base_idx = chunk * BLOCK_SIZE + tid;

        // Load 4 elements at once
        b_vec[chunk] = reinterpret_cast<aligned_bf16x4 const*>(B)[row * GEMM_K / VEC_SIZE + base_idx];
    }
    asm volatile("griddepcontrol.wait;" ::: "memory");

    // Process 5 chunks of 4 elements each
#pragma unroll
    for (int chunk = 0; chunk < GEMM_K / BLOCK_SIZE / VEC_SIZE; chunk++)
    {
        // Base index for this chunk
        int base_idx = chunk * BLOCK_SIZE + tid;

        // Load 4 elements at once
        aligned_bf16x4 a_vec = reinterpret_cast<aligned_bf16x4 const*>(A)[token_idx * GEMM_K / VEC_SIZE + base_idx];
#pragma unroll
        for (int i = 0; i < VEC_SIZE; i += 2)
        {

            float2 a_val = make_float2(a_vec.data[i], a_vec.data[i + 1]);
            float2 b_val = make_float2(b_vec[chunk].data[i], b_vec[chunk].data[i + 1]);

#if __CUDA_ARCH__ >= 1000
            thread_sum = __ffma2_rn(a_val, b_val, thread_sum);
#else
            thread_sum.x += a_val.x * b_val.x;
            thread_sum.y += a_val.y * b_val.y;
#endif
        }
    }

    // Warp-level reduction
    float warp_sum = thread_sum.x + thread_sum.y;
    for (int offset = warpSize / 2; offset > 0; offset >>= 1)
    {
        warp_sum += __shfl_down_sync(0xffffffff, warp_sum, offset);
    }

    // First thread in each warp writes to shared memory
    if (tid % warpSize == 0)
    {
        reduce_buffer[tid / warpSize] = warp_sum;
    }
    __syncthreads();

    // Final thread reduces across warps and writes the result
    if (tid == 0)
    {
        float block_sum = 0.0f;
        for (int i = 0; i < BLOCK_SIZE / warpSize; i++)
        {
            block_sum += reduce_buffer[i];
        }
        C[token_idx * NUM_EXPERTS + row] = __float2bfloat16(block_sum);
    }
#endif
}

// Function to launch kernel using FDL (Flexible Dispatch Layer)
void launch_kernel_fdl(
    dim3 grid_dim, dim3 block_dim, hipStream_t stream, void* kernel_func, void* args[], int num_args)
{
    hipLaunchConfig_t config;
    config.gridDim = grid_dim;
    config.blockDim = block_dim;
    config.dynamicSmemBytes = 0;
    config.stream = stream;

    hipLaunchAttribute attrs[1];
    config.attrs = attrs;
    config.numAttrs = 0;
    attrs[config.numAttrs].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[config.numAttrs++].val.programmaticStreamSerializationAllowed = 1;

    hipLaunchKernelExC(&config, (void const*) kernel_func, args);
}

void gemv_kernel_launcher(
    int num_tokens, __hip_bfloat16 const* A, __hip_bfloat16 const* B, __hip_bfloat16* C, hipStream_t stream)
{

    int const grid_size = NUM_EXPERTS * num_tokens;

    void* args[] = {(void*) &num_tokens, (void*) &A, (void*) &B, (void*) &C};
    launch_kernel_fdl(dim3(grid_size), dim3(BLOCK_SIZE), stream, (void*) gemv_kernel, args, 4);
}

void llama4_router_gemm_op(int num_tokens, void const* A, void const* B, void* C, hipStream_t stream)
{
    __hip_bfloat16 const* A_bf16 = static_cast<__hip_bfloat16 const*>(A);
    __hip_bfloat16 const* B_bf16 = static_cast<__hip_bfloat16 const*>(B);
    __hip_bfloat16* C_bf16 = static_cast<__hip_bfloat16*>(C);

    gemv_kernel_launcher(num_tokens, A_bf16, B_bf16, C_bf16, stream);
}

} // namespace tensorrt_llm::kernels::llama4_router_gemm
