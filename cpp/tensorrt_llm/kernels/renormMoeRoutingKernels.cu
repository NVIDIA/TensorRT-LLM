#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/kernels/archCondition.h"
#include "tensorrt_llm/kernels/renormMoeRoutingKernels.h"
#include <climits> // For INT_MAX
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <cuda/std/limits> // For numeric_limits
#include <math.h>

namespace cg = cooperative_groups;
using namespace tensorrt_llm::common;

namespace tensorrt_llm::kernels
{

static constexpr int BLOCK_SIZE = 1024;
static constexpr int WARP_SIZE = 32;
static constexpr int WARPS_PER_BLOCK = BLOCK_SIZE / WARP_SIZE;

namespace reduce_topk
{

static constexpr bool TLLM_GEN_HAS_FAST_REDUX = tensorrt_llm::kernels::arch::is_major_v<10>;

template <typename T_>
struct TopKRedType
{
    using T = T_;
    static_assert(std::is_same_v<T, float> || std::is_same_v<T, half> || std::is_same_v<T, __hip_bfloat16>,
        "Top K reduction only implemented for float, float16 and bfloat16");

    using TypeCmp = std::conditional_t<sizeof(T) == 4, uint64_t, uint32_t>;
    using IdxT = std::conditional_t<sizeof(T) == 4, int32_t, int16_t>;
    static constexpr int moveBits = (sizeof(T) == 4) ? 32 : 16;
    static constexpr int maxIdx = 65535;
    TypeCmp compValIdx;

    static __host__ __device__ inline TypeCmp makeCmpVal(T val, int32_t idx = 0)
    {
        auto valueBits = hipcub::Traits<T>::TwiddleIn(reinterpret_cast<typename hipcub::Traits<T>::UnsignedBits&>(val));
        TypeCmp compactTmp = reinterpret_cast<TypeCmp&>(valueBits);
        compactTmp = (compactTmp << moveBits) | (0xFFFF & (maxIdx - idx));
        // Use 65535 minus idx to give higher priority to elements with smaller indices.
        return compactTmp;
    }

    static __host__ __device__ void unpack(T& value, int32_t& index, TypeCmp cmp)
    {
        // Since “65535-idx” is always smaller than 65536 and positive, we can directly use it as the lower 16 bits
        index = maxIdx - static_cast<int32_t>((cmp & 0xFFFF));

        auto compactTmp = cmp >> moveBits;
        auto valueBits
            = hipcub::Traits<T>::TwiddleOut(reinterpret_cast<typename hipcub::Traits<T>::UnsignedBits&>(compactTmp));
        value = reinterpret_cast<T&>(valueBits);
    }

    __host__ __device__ TopKRedType() = default;

    __host__ __device__ TopKRedType(T val, int32_t idx)
        : compValIdx(makeCmpVal(val, idx))
    {
    }

    __host__ __device__ operator TypeCmp() const noexcept
    {
        return compValIdx;
    }

    __device__ inline TypeCmp reduce(cg::thread_block_tile<WARP_SIZE> const& warp)
    {
        if constexpr (!TLLM_GEN_HAS_FAST_REDUX || sizeof(TypeCmp) == 8)
        {
            return cg::reduce(warp, compValIdx, cg::greater<TypeCmp>{});
        }
        else
        {
            TypeCmp result;
            asm("redux.sync.max.u32 %0, %1, 0xffffffff;\n" : "=r"(result) : "r"(compValIdx));
            return result;
        }
    }
};

////////////////////////////////////////////////////////////////////////////////////////////////////

template <int K_, bool Enable_>
struct TopKIdx
{
    // by default, empty
};

template <int K_>
struct TopKIdx<K_, true>
{
    static constexpr int K = K_;
    int32_t val[K];
};

////////////////////////////////////////////////////////////////////////////////////////////////////

#define TOPK_SWAP(I, J)                                                                                                \
    {                                                                                                                  \
        auto pairMin = min(topK[I].compValIdx, topK[J].compValIdx);                                                    \
        auto pairMax = max(topK[I].compValIdx, topK[J].compValIdx);                                                    \
        topK[I].compValIdx = pairMax;                                                                                  \
        topK[J].compValIdx = pairMin;                                                                                  \
    }

template <int N, typename RedType>
struct Sort;

template <typename RedType>
struct Sort<1, RedType>
{
    static __device__ void run(RedType* topK) {}
};

template <typename RedType>
struct Sort<2, RedType>
{
    static __device__ void run(RedType* topK)
    {
        TOPK_SWAP(0, 1);
    }
};

template <typename RedType>
struct Sort<3, RedType>
{
    static __device__ void run(RedType* topK)
    {
        TOPK_SWAP(0, 1);
        TOPK_SWAP(1, 2);
        TOPK_SWAP(0, 1);
    }
};

template <typename RedType>
struct Sort<4, RedType>
{
    static __device__ void run(RedType* topK)
    {
        TOPK_SWAP(0, 2);
        TOPK_SWAP(1, 3);
        TOPK_SWAP(0, 1);
        TOPK_SWAP(2, 3);
        TOPK_SWAP(1, 2);
    }
};

template <int K, typename Type, int N, bool IsSorted = false>
__device__ void reduceTopK(cg::thread_block_tile<WARP_SIZE> const& warp, Type (&out)[K], int32_t (&outIdx)[K],
    Type (&value)[N], int32_t (&idx)[N], Type minValue)
{
    static_assert(K > 0, "Top K must have K > 0");
    static_assert(K < WARP_SIZE, "Top K must have K < WARP_SIZE");
    static_assert(N > 0, "Top K must have N > 0");
    static_assert(N < 5, "Only support candidates number less than or equal to 128");
    using RedType = TopKRedType<Type>;
    RedType topK[N];
#pragma unroll
    for (int nn = 0; nn < N; ++nn)
    {
        topK[nn] = RedType{value[nn], idx[nn]};
    }

    if constexpr (!IsSorted)
    {
        Sort<N, RedType>::run(topK);
    }
    typename RedType::TypeCmp packedMax{};
#pragma unroll
    for (int kk = 0; kk < K; ++kk)
    {
        bool update = kk > 0 && packedMax == topK[0].compValIdx;
#pragma unroll
        for (int nn = 0; nn < N; ++nn)
        {
            topK[nn] = update && nn == N - 1 ? RedType{minValue, idx[nn]} : update ? topK[nn + 1] : topK[nn];
        }
        // get the next largest value
        packedMax = topK[0].reduce(warp);
        RedType::unpack(out[kk], outIdx[kk], packedMax);
    }
};

#undef TOPK_SWAP

} // end of namespace reduce_topk

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__device__ T calcSoftmax(cg::thread_block_tile<WARP_SIZE> const& warp, T score, int32_t laneIdx, int32_t NumTopExperts)
{
    T maxScore = T{-INFINITY};
    if (laneIdx < NumTopExperts)
    {
        maxScore = score >= maxScore ? score : maxScore;
    }
    maxScore = cg::reduce(warp, maxScore, cg::greater<T>());

    float sumScore = float{0.f};
    float newScore;
    // Get the summation of scores for each token
    if (laneIdx < NumTopExperts)
    {
        newScore = static_cast<float>(score) - static_cast<float>(maxScore);
        newScore = static_cast<float>(exp(newScore));
        sumScore += newScore;
    }
    sumScore = cg::reduce(warp, sumScore, cg::plus<float>());

    if (laneIdx < NumTopExperts)
    {
        score = static_cast<T>(newScore / sumScore);
    }

    return score;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename InputT, typename OutputT, typename IdxT, int MaxNumExperts, int MaxNumTopExperts>
__global__ void renormMoeRoutingKernel(InputT* routerLogits, OutputT* topkValues, IdxT* topkIndices,
    int32_t const numTokens, int32_t const numExperts, int32_t const topK)
{

    uint32_t const blockRank = blockIdx.x;
    uint32_t const tIdx = BLOCK_SIZE * blockRank + threadIdx.x;
    uint32_t const warpIdx = tIdx / WARP_SIZE;
    uint32_t const laneIdx = tIdx % WARP_SIZE;
    uint32_t const warpNum = gridDim.x * WARPS_PER_BLOCK;
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WARP_SIZE>(block);

    InputT minScore = InputT{-INFINITY};
    for (uint32_t tokenId = warpIdx; tokenId < numTokens; tokenId += warpNum)
    {
        auto scoreOffset = tokenId * numExperts;
        auto outputOffset = tokenId * topK;
        InputT inputScore[MaxNumExperts / WARP_SIZE];
        IdxT inputIndex[MaxNumExperts / WARP_SIZE];

        InputT warpTopKScore[MaxNumTopExperts];
        IdxT warpTopKExpertIdx[MaxNumTopExperts];

        // Load scores and indices for this warp
        for (uint32_t i = 0; i < MaxNumExperts / WARP_SIZE; ++i)
        {
            auto expertIdx = i * WARP_SIZE + laneIdx;
            inputScore[i]
                = expertIdx < numExperts ? static_cast<InputT>(routerLogits[scoreOffset + expertIdx]) : minScore;
            inputIndex[i] = expertIdx;
        }

        // Reduce topK scores and indices for this warp
        reduce_topk::reduceTopK(warp, warpTopKScore, warpTopKExpertIdx, inputScore, inputIndex, minScore);

        // Perform softmax on topK scores
        auto score = calcSoftmax(warp,
            laneIdx < topK ? static_cast<float>(warpTopKScore[laneIdx]) : static_cast<float>(minScore), laneIdx, topK);
        if (laneIdx < topK)
        {
            topkValues[outputOffset + laneIdx] = static_cast<OutputT>(score);
            topkIndices[outputOffset + laneIdx] = warpTopKExpertIdx[laneIdx];
        }
    } // end for tokenId
}

int nextPowerOfTwo(int num)
{
    if (num <= 0)
    {
        return 1; // Handle invalid input
    }
    int power = 1;
    while (power < num)
    {
        // Check for overflow before shifting
        if (power > INT_MAX / 2)
        {
            return power;
        }
        power <<= 1;
    }
    return power;
}

#define CASE(MAX_NUM_EXPERTS)                                                                                          \
    case MAX_NUM_EXPERTS:                                                                                              \
        switch (maxNumTopExperts)                                                                                      \
        {                                                                                                              \
        case 1: kernelInstance = &renormMoeRoutingKernel<InputT, OutputT, IdxT, MAX_NUM_EXPERTS, 1>; break;            \
        case 2: kernelInstance = &renormMoeRoutingKernel<InputT, OutputT, IdxT, MAX_NUM_EXPERTS, 2>; break;            \
        case 4: kernelInstance = &renormMoeRoutingKernel<InputT, OutputT, IdxT, MAX_NUM_EXPERTS, 4>; break;            \
        case 8: kernelInstance = &renormMoeRoutingKernel<InputT, OutputT, IdxT, MAX_NUM_EXPERTS, 8>; break;            \
        default: kernelInstance = nullptr; break;                                                                      \
        }                                                                                                              \
        break;

template <typename InputT, typename OutputT, typename IdxT>
void invokeRenormMoeRouting(InputT* routerLogits, OutputT* topkValues, IdxT* topkIndices, int64_t const numTokens,
    int64_t const numExperts, int64_t const topK, hipStream_t const stream)
{

    const uint32_t maxNumBlocks = 1024;
    const uint32_t numBlocks = std::min(static_cast<uint32_t>((numTokens - 1) / WARPS_PER_BLOCK + 1), maxNumBlocks);

    uint32_t maxNumExperts = nextPowerOfTwo(numExperts) < 32 ? 32 : nextPowerOfTwo(numExperts);
    uint32_t maxNumTopExperts = nextPowerOfTwo(topK);

    auto* kernelInstance = &renormMoeRoutingKernel<InputT, OutputT, IdxT, 128, 8>;

    switch (maxNumExperts)
    {
        CASE(32)
        CASE(64)
        CASE(96)
        CASE(128)
    default: kernelInstance = nullptr; break;
    }

    if (kernelInstance == nullptr)
    {
        TLLM_CHECK_WITH_INFO(kernelInstance != nullptr, "Can not find corresponding kernel instance.");
    }

    dim3 renormMoeRoutingGridDim(numBlocks);
    dim3 renormMoeRoutingBlockDim(BLOCK_SIZE);
    hipLaunchConfig_t config;
    config.gridDim = renormMoeRoutingGridDim;
    config.blockDim = renormMoeRoutingBlockDim;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    cudaLaunchKernelEx(&config, kernelInstance, routerLogits, topkValues, topkIndices, static_cast<int32_t>(numTokens),
        static_cast<int32_t>(numExperts), static_cast<int32_t>(topK));
    sync_check_cuda_error(stream);
}

#define INSTANTIATE_RENORM_MOE_ROUTING(InputT, OutputT, IdxT)                                                          \
    template void invokeRenormMoeRouting<InputT, OutputT, IdxT>(InputT * routerLogits, OutputT * topkValues,           \
        IdxT * topkIndices, int64_t const numTokens, int64_t const numExperts, int64_t const topK,                     \
        hipStream_t const stream);

INSTANTIATE_RENORM_MOE_ROUTING(float, float, int32_t);
INSTANTIATE_RENORM_MOE_ROUTING(half, float, int32_t);
#ifdef ENABLE_BF16
INSTANTIATE_RENORM_MOE_ROUTING(__hip_bfloat16, float, int32_t);
#endif

} // namespace tensorrt_llm::kernels
