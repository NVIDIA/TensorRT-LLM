#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "unfusedAttentionKernels_2_template.h"

namespace tensorrt_llm
{
namespace kernels
{

#ifdef ENABLE_FP4
INSTANTIATE_ATTENTION_INPUT_PROCESSING(half, __hip_fp4_e2m1, KVBlockArray);
INSTANTIATE_ATTENTION_INPUT_PROCESSING(half, __hip_fp4_e2m1, KVLinearBuffer);
#endif

} // namespace kernels
} // namespace tensorrt_llm
