#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/kernels/noAuxTcKernels.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;
using namespace tensorrt_llm::common;

namespace tensorrt_llm::kernels
{
constexpr unsigned FULL_WARP_MASK = 0xffffffff;
constexpr int32_t WARP_SIZE = 32;
constexpr int32_t BLOCK_SIZE = 512;
constexpr int32_t NUM_WARPS_PER_BLOCK = BLOCK_SIZE / WARP_SIZE;

namespace warp_topk
{

template <int size, typename T>
__host__ __device__ constexpr T round_up_to_multiple_of(T len)
{
    if (len == 0)
    {
        return 0;
    }
    return ((len - 1) / size + 1) * size;
}

template <typename T>
constexpr __host__ __device__ bool isPowerOf2(T v)
{
    return (v && !(v & (v - 1)));
}

template <bool greater, typename T>
__device__ bool is_better_than(T val, T baseline)
{
    return (val > baseline && greater) || (val < baseline && !greater);
}

template <typename T, typename idxT>
int calc_smem_size_for_block_wide(int num_of_warp, int64_t k)
{
    int64_t cache_topk = (sizeof(T) + sizeof(idxT)) * num_of_warp * k;
    int64_t n = std::max<int>(num_of_warp / 2 * k, num_of_warp * WARP_SIZE);
    return max(cache_topk, round_up_to_multiple_of<256>(n * sizeof(T)) + n * sizeof(idxT));
}

template <int size, bool ascending, typename T, typename idxT>
struct BitonicMerge
{
    // input should be a bitonic sequence, and sort it to be a monotonic sequence
    __device__ static void merge(T* __restrict__ val_arr, idxT* __restrict__ idx_arr)
    {
        static_assert(isPowerOf2(size));
        static_assert(size >= 2 * WARP_SIZE);
        constexpr int arr_len = size / WARP_SIZE;

        constexpr int stride = arr_len / 2;
        for (int i = 0; i < stride; ++i)
        {
            int const other_i = i + stride;
            T& val = val_arr[i];
            T& other_val = val_arr[other_i];
            if ((val > other_val && ascending) || (val < other_val && !ascending))
            {
                T tmp = val;
                val = other_val;
                other_val = tmp;

                idxT tmp2 = idx_arr[i];
                idx_arr[i] = idx_arr[other_i];
                idx_arr[other_i] = tmp2;
            }
        }

        BitonicMerge<size / 2, ascending, T, idxT>::merge(val_arr, idx_arr);
        BitonicMerge<size / 2, ascending, T, idxT>::merge(val_arr + arr_len / 2, idx_arr + arr_len / 2);
    }
};

template <int size, bool ascending, typename T, typename idxT>
struct BitonicSort
{
    __device__ static void sort(T* __restrict__ val_arr, idxT* __restrict__ idx_arr)
    {
        static_assert(isPowerOf2(size));
        static_assert(size >= 2 * WARP_SIZE);
        constexpr int arr_len = size / WARP_SIZE;

        BitonicSort<size / 2, true, T, idxT>::sort(val_arr, idx_arr);
        BitonicSort<size / 2, false, T, idxT>::sort(val_arr + arr_len / 2, idx_arr + arr_len / 2);
        BitonicMerge<size, ascending, T, idxT>::merge(val_arr, idx_arr);
    }
};

template <bool ascending, typename T, typename idxT>
struct BitonicSort<32, ascending, T, idxT>
{
    __device__ static void sort(T* __restrict__ val_arr, idxT* __restrict__ idx_arr)
    {
        int const lane = threadIdx.x % WARP_SIZE;

        // ascending doesn't matter before merging since all we need is a bitonic sequence
        for (int stage = 0; stage < 4; ++stage)
        {
            for (int stride = (1 << stage); stride > 0; stride /= 2)
            {
                bool reverse = (lane >> stage) & 2;
                bool is_second = lane & stride;

                T other = __shfl_xor_sync(FULL_WARP_MASK, *val_arr, stride);
                idxT other_idx = __shfl_xor_sync(FULL_WARP_MASK, *idx_arr, stride);
                if (*val_arr != other && (*val_arr > other) != (reverse != is_second))
                {
                    *val_arr = other;
                    *idx_arr = other_idx;
                }
            }
        }

        BitonicMerge<32, ascending, T, idxT>::merge(val_arr, idx_arr);
    }
};

template <bool ascending, typename T, typename idxT>
struct BitonicMerge<32, ascending, T, idxT>
{
    __device__ static void merge(T* __restrict__ val_arr, idxT* __restrict__ idx_arr)
    {
        int const lane = threadIdx.x % WARP_SIZE;
        for (int stride = WARP_SIZE / 2; stride > 0; stride /= 2)
        {
            bool is_second = lane & stride;
            T& val = *val_arr;
            T other = __shfl_xor_sync(FULL_WARP_MASK, val, stride);
            idxT& idx = *idx_arr;
            idxT other_idx = __shfl_xor_sync(FULL_WARP_MASK, idx, stride);
            if (val != other && ((val > other) == (ascending != is_second)))
            {
                val = other;
                idx = other_idx;
            }
        }
    }
};

template <int capacity, bool greater, typename T, typename idxT>
class WarpSort
{
public:
    __device__ WarpSort(idxT k, T dummy)
        : lane_(threadIdx.x % WARP_SIZE)
        , k_(k)
        , dummy_(dummy)
    {
        static_assert(capacity >= WARP_SIZE && isPowerOf2(capacity));

        for (int i = 0; i < max_arr_len_; ++i)
        {
            val_arr_[i] = dummy_;
            idx_arr_[i] = 0;
        }
    }

    // load and merge k sorted values
    __device__ void load_sorted(T const* __restrict__ in, idxT const* __restrict__ in_idx, idxT start)
    {
        idxT idx = start + WARP_SIZE - 1 - lane_;
        for (int i = max_arr_len_ - 1; i >= 0; --i, idx += WARP_SIZE)
        {
            if (idx < start + k_)
            {
                T t = in[idx];
                if (is_better_than<greater>(t, val_arr_[i]))
                {
                    val_arr_[i] = t;
                    idx_arr_[i] = in_idx[idx];
                }
            }
        }

        BitonicMerge<capacity, !greater, T, idxT>::merge(val_arr_, idx_arr_);
    }

    __device__ void dump(T* __restrict__ out, idxT* __restrict__ out_idx) const
    {
        for (int i = 0; i < max_arr_len_; ++i)
        {
            idxT out_i = i * WARP_SIZE + lane_;
            if (out_i < k_)
            {
                out[out_i] = val_arr_[i];
                out_idx[out_i] = idx_arr_[i];
            }
        }
    }

    __device__ void dumpIdx(idxT* __restrict__ out_idx) const
    {
        for (int i = 0; i < max_arr_len_; ++i)
        {
            idxT out_i = i * WARP_SIZE + lane_;
            if (out_i < k_)
            {
                out_idx[out_i] = idx_arr_[i];
            }
        }
    }

protected:
    static constexpr int max_arr_len_ = capacity / WARP_SIZE;

    T val_arr_[max_arr_len_];
    idxT idx_arr_[max_arr_len_];

    int const lane_;
    idxT const k_;
    T const dummy_;

}; // end class WarpSort

template <int capacity, bool greater, typename T, typename idxT>
class WarpSelect : public WarpSort<capacity, greater, T, idxT>
{
public:
    __device__ WarpSelect(idxT k, T dummy)
        : WarpSort<capacity, greater, T, idxT>(k, dummy)
        , k_th_(dummy)
        , k_th_lane_((k - 1) % WARP_SIZE)
    {

        extern __shared__ char smem_buf[]; // extern __shared__ T smem_buf[];

        int const num_of_warp = blockDim.x / WARP_SIZE;
        int const warp_id = threadIdx.x / WARP_SIZE;
        val_smem_ = reinterpret_cast<T*>(smem_buf);
        val_smem_ += warp_id * WARP_SIZE;
        idx_smem_
            = reinterpret_cast<idxT*>(smem_buf + round_up_to_multiple_of<256>(num_of_warp * sizeof(T) * WARP_SIZE));
        idx_smem_ += warp_id * WARP_SIZE;
    }

    __device__ void add(T const* in, idxT start, idxT end)
    {
        idxT const end_for_fullwarp = round_up_to_multiple_of<WARP_SIZE>(end - start) + start;
        for (idxT i = start + lane_; i < end_for_fullwarp; i += WARP_SIZE)
        {
            T val = (i < end) ? in[i] : dummy_;
            add(val, i);
        }
    }

    __device__ void add(T val, idxT idx)
    {
        bool do_add = is_better_than<greater>(val, k_th_);
        uint32_t mask = __ballot_sync(FULL_WARP_MASK, do_add);
        if (mask == 0)
        {
            return;
        }

        int pos = smem_buf_len_ + __popc(mask & ((0x1u << lane_) - 1));
        if (do_add && pos < WARP_SIZE)
        {
            val_smem_[pos] = val;
            idx_smem_[pos] = idx;
            do_add = false;
        }
        smem_buf_len_ += __popc(mask);
        if (smem_buf_len_ >= WARP_SIZE)
        {
            __syncwarp();
            merge_buf_(val_smem_[lane_], idx_smem_[lane_]);
            smem_buf_len_ -= WARP_SIZE;
        }
        if (do_add)
        {
            pos -= WARP_SIZE;
            val_smem_[pos] = val;
            idx_smem_[pos] = idx;
        }
        __syncwarp();
    }

    __device__ void done()
    {
        if (smem_buf_len_)
        {
            T val = (lane_ < smem_buf_len_) ? val_smem_[lane_] : dummy_;
            idxT idx = (lane_ < smem_buf_len_) ? idx_smem_[lane_] : 0;
            merge_buf_(val, idx);
        }

        // after done(), smem is used for merging results among warps
        __syncthreads();
    }

private:
    __device__ void set_k_th_()
    {
        k_th_ = __shfl_sync(FULL_WARP_MASK, val_arr_[max_arr_len_ - 1], k_th_lane_);
    }

    __device__ void merge_buf_(T val, idxT idx)
    {
        BitonicSort<WARP_SIZE, greater, T, idxT>::sort(&val, &idx);

        T& old = val_arr_[max_arr_len_ - 1];
        if (is_better_than<greater>(val, old))
        {
            old = val;
            idx_arr_[max_arr_len_ - 1] = idx;
        }

        BitonicMerge<capacity, !greater, T, idxT>::merge(val_arr_, idx_arr_);

        set_k_th_();
    }

    using WarpSort<capacity, greater, T, idxT>::max_arr_len_;
    using WarpSort<capacity, greater, T, idxT>::val_arr_;
    using WarpSort<capacity, greater, T, idxT>::idx_arr_;
    using WarpSort<capacity, greater, T, idxT>::lane_;
    using WarpSort<capacity, greater, T, idxT>::k_;
    using WarpSort<capacity, greater, T, idxT>::dummy_;

    T* val_smem_;
    idxT* idx_smem_;
    int smem_buf_len_ = 0;

    T k_th_;
    int const k_th_lane_;
}; // end class WarpSelect
} // namespace warp_topk

template <typename T>
__device__ void topk_with_k2(T* output, T const* input, cg::thread_block_tile<32> const& tile, int32_t const lane_id,
    int const num_experts_per_group)
{
    // Get the top2 per thread
    T largest = -INFINITY;
    T second_largest = -INFINITY;

    if (num_experts_per_group > WARP_SIZE)
    {
        for (int i = lane_id; i < num_experts_per_group; i += WARP_SIZE)
        {
            T value = input[i];
            if (value > largest)
            {
                second_largest = largest;
                largest = value;
            }
            else if (value > second_largest)
            {
                second_largest = value;
            }
        }
    }
    else
    {
        for (int i = lane_id; i < num_experts_per_group; i += WARP_SIZE)
        {
            largest = input[i];
        }
    }

    __syncwarp(); // Ensure all threads have valid data before reduction
    // Get the top2 warpwise
    T max1 = cg::reduce(tile, largest, cg::greater<T>());

    T max2 = max1;
    bool equal_to_max1 = (max1 == largest);

    int count_max1 = __popc(__ballot_sync(FULL_WARP_MASK, equal_to_max1));

    if (count_max1 == 1)
    {
        largest = (largest == max1) ? second_largest : largest;
        max2 = cg::reduce(tile, largest, cg::greater<T>());
    }

    if (lane_id == 0)
    {
        *output = max1 + max2;
    }
}

template <typename T>
__global__ void topk_with_k2_kernel(T* output, T* input, int64_t const num_tokens, int64_t const num_cases,
    int64_t const n_group, int64_t const num_experts_per_group)
{

    int32_t warp_id = threadIdx.x / WARP_SIZE;
    int32_t lane_id = threadIdx.x % WARP_SIZE;

    int32_t case_id = blockIdx.x * NUM_WARPS_PER_BLOCK + warp_id;
    if (case_id < num_cases)
    {
        input += case_id * num_experts_per_group;
        output += case_id;

        cg::thread_block block = cg::this_thread_block();
        cg::thread_block_tile<32> tile = cg::tiled_partition<32>(block);

        topk_with_k2(output, input, tile, lane_id, num_experts_per_group);
    }
}

template <typename T>
__global__ void group_idx_and_topk_idx_kernel(T* scores, T const* group_scores, T* scores_with_bias,
    int64_t const num_tokens, int64_t const n_group, int64_t const topk_group, int64_t const topk,
    int64_t const num_experts, int64_t const num_experts_per_group, double routed_scaling_factor)
{
    int32_t warp_id = threadIdx.x / WARP_SIZE;
    int32_t lane_id = threadIdx.x % WARP_SIZE;
    int32_t case_id = blockIdx.x * NUM_WARPS_PER_BLOCK + warp_id; // one per token
    scores_with_bias += case_id * num_experts;
    scores += case_id * num_experts;
    group_scores += case_id * n_group;
    int32_t align_num_experts_per_group = warp_topk::round_up_to_multiple_of<WARP_SIZE>(num_experts_per_group);

    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> tile = cg::tiled_partition<32>(block);

    extern __shared__ char smem_buf[]; // NOTE: reuse the shared memory here to store the target topk idx
    int32_t* s_topk_idx = reinterpret_cast<int32_t*>(smem_buf);
    T* s_topk_value = reinterpret_cast<T*>(s_topk_idx + NUM_WARPS_PER_BLOCK * topk) + warp_id * topk;
    s_topk_idx += warp_id * topk;

    T value = cuda::std::numeric_limits<T>::min();
    T topk_group_value = cuda::std::numeric_limits<T>::min();
    int32_t num_equalto_topkth_group;

    if (case_id < num_tokens)
    {
        // calculate group_idx
        int32_t target_num_min = WARP_SIZE - n_group + topk_group;
        if (lane_id < n_group
            && (isfinite(cuda_cast<float, T>(group_scores[lane_id])))) // The check is necessary to avoid abnormal input
        {
            value = group_scores[lane_id];
        }

        int count_equal_to_top_value = WARP_SIZE - n_group;
        int pre_count_equal_to_top_value = 0;
        // Use loop to find the largset top_group
        while (count_equal_to_top_value < target_num_min)
        {
            __syncwarp(); // Ensure all threads have valid data before reduction
            topk_group_value = cg::reduce(tile, value, cg::greater<T>());
            if (value == topk_group_value)
            {
                value = cuda::std::numeric_limits<T>::min();
            }
            pre_count_equal_to_top_value = count_equal_to_top_value;
            count_equal_to_top_value
                = __popc(__ballot_sync(FULL_WARP_MASK, (value == cuda::std::numeric_limits<T>::min())));
        }
        num_equalto_topkth_group = target_num_min - pre_count_equal_to_top_value;
    }
    __syncthreads();

    warp_topk::WarpSelect</*capability*/ WARP_SIZE, /*greater*/ true, T, int32_t> queue((int32_t) topk, -INFINITY);

    int count_equalto_topkth_group = 0;
    bool if_proceed_next_topk = (topk_group_value != cuda::std::numeric_limits<T>::min());
    if (case_id < num_tokens && if_proceed_next_topk)
    {
        for (int i_group = 0; i_group < n_group; i_group++)
        {
            if ((group_scores[i_group] > topk_group_value)
                || ((group_scores[i_group] == topk_group_value)
                    && (count_equalto_topkth_group < num_equalto_topkth_group)))
            {
                int32_t offset = i_group * num_experts_per_group;
                for (int32_t i = lane_id; i < align_num_experts_per_group; i += WARP_SIZE)
                {
                    T candidates
                        = (i < num_experts_per_group) && isfinite(cuda_cast<float, T>(scores_with_bias[offset + i]))
                        ? scores_with_bias[offset + i]
                        : cuda::std::numeric_limits<T>::min();
                    queue.add(candidates, offset + i);
                }
                if (group_scores[i_group] == topk_group_value)
                {
                    count_equalto_topkth_group++;
                }
            }
        }
        queue.done();
        __syncwarp();
        // Get the topk_idx
        queue.dumpIdx(s_topk_idx);
        __syncwarp();
    }

    // Load the valid score value
    // Calculate the summation
    float topk_sum = 1e-20;
    if (case_id < num_tokens && if_proceed_next_topk)
    {
        for (int i = lane_id; i < warp_topk::round_up_to_multiple_of<WARP_SIZE>(topk); i += WARP_SIZE)
        {
            T value = i < topk ? scores[s_topk_idx[i]] : cuda_cast<T, float>(0.0f); // Load the valid value of expert
            if (i < topk)
            {
                s_topk_value[i] = value;
            }
            topk_sum += reduce(tile, cuda_cast<float, T>(value), cg::plus<float>());
        }
    }

    __syncthreads();

    if (case_id < num_tokens)
    {
        if (if_proceed_next_topk)
        {
            if (case_id < num_tokens)
            {
                for (int i = lane_id; i < num_experts; i += WARP_SIZE)
                {
                    scores[i] = 0;
                }
            }
            __threadfence();
            __syncthreads();
            for (int i = lane_id; i < topk; i += WARP_SIZE)
            {
                float value = cuda_cast<float, T>(s_topk_value[i]) / topk_sum * routed_scaling_factor;
                scores[s_topk_idx[i]] = cuda_cast<T, float>(value);
            }
        }
        else
        {
            for (int i = lane_id; i < num_experts; i += WARP_SIZE)
            {
                scores[i] = i < topk ? cuda_cast<T, float>(1.0f / topk) : cuda_cast<T, float>(0.0f);
            }
        }
        // Note: when if_proceed_next_topk==false, choose the first 8 experts as the default result.
        //@TODO: check if this default strategy is acceptable. Might need to leave it as nan array.
    }
}

template <typename T>
void invokeNoAuxTc(T* scores, T* group_scores, T* scores_with_bias, int64_t const num_tokens, int64_t const num_experts,
    int64_t const n_group, int64_t const topk_group, int64_t const topk, double const routed_scaling_factor,
    hipStream_t const stream)
{
    int64_t num_cases = num_tokens * n_group;
    int64_t topk_with_k2_num_blocks = (num_cases - 1) / NUM_WARPS_PER_BLOCK + 1;
    topk_with_k2_kernel<T><<<topk_with_k2_num_blocks, BLOCK_SIZE, 0, stream>>>(
        group_scores, scores_with_bias, num_tokens, num_cases, n_group, num_experts / n_group);
    sync_check_cuda_error();
    int64_t topk_with_k_group_num_blocks = (num_tokens - 1) / NUM_WARPS_PER_BLOCK + 1;
    size_t dynamic_smem_in_bytes = warp_topk::calc_smem_size_for_block_wide<T, int32_t>(NUM_WARPS_PER_BLOCK, topk);

    group_idx_and_topk_idx_kernel<T><<<topk_with_k_group_num_blocks, BLOCK_SIZE, dynamic_smem_in_bytes, stream>>>(
        scores, group_scores, scores_with_bias, num_tokens, n_group, topk_group, topk, num_experts,
        num_experts / n_group, routed_scaling_factor);
    sync_check_cuda_error();
}

#define INSTANTIATE_NOAUX_TC(T)                                                                                        \
    template void invokeNoAuxTc<T>(T * scores, T * group_scores, T * scores_with_bias, int64_t const num_tokens,       \
        int64_t const num_experts, int64_t const n_group, int64_t const topk_group, int64_t const topk,                \
        double const routed_scaling_factor, hipStream_t const stream);

INSTANTIATE_NOAUX_TC(float);
INSTANTIATE_NOAUX_TC(half);
#ifdef ENABLE_BF16
INSTANTIATE_NOAUX_TC(__hip_bfloat16);
#endif
} // namespace tensorrt_llm::kernels
