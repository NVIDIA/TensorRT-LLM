#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "moeTopKFuncs.cuh"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/kernels/noAuxTcKernels.h"
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;
using namespace tensorrt_llm::common;

namespace tensorrt_llm::kernels
{
static constexpr int WARP_SIZE = 32;
static constexpr int NumKimiK2Experts = 384;
static constexpr int NumDeepseekExperts = 256;
static constexpr int MaxNumExpertsUnit = 128;
static constexpr int NumTopGroupScores = 2;
static constexpr int MaxNumTopExperts = 8;
static constexpr int MaxNumTopGroups = 4;

static __device__ inline float sigmoid_accurate(float x)
{
    return 0.5f * tanhf(0.5f * x) + 0.5f;
}

template <typename InputT, typename BiasT, typename OutputT, typename IdxT, int MaxNumExperts, bool UseGroups>
__global__ void deepseek_v3_topk_kernel(InputT* scores, OutputT* topkValues, IdxT* topkIndices, BiasT* routingBias,
    int64_t const numTokens, int64_t const numGroup, int64_t const topkGroup, int64_t const topk,
    int64_t const numExperts, int64_t const numExpertsPerGroup, double const routedScalingFactor)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

    // declare shared memory structure
    // number of experts is bounded by number of threads
    __shared__ float __attribute((aligned(128))) smemScoreSigmoid[MaxNumExperts];
    __shared__ float __attribute((aligned(128))) smemScoreBias[MaxNumExperts];
    // number of expert groups is bounded by number of warps
    int constexpr NumWarps = MaxNumExperts / WARP_SIZE;
    __shared__ float __attribute((aligned(128))) smemGroupScores[NumWarps];

    // needed for warp reduce
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WARP_SIZE>(block);

    // for the final reduction of weight norm, only some lanes need to participate
    int32_t laneIdx = threadIdx.x % WARP_SIZE;
    int32_t warpIdx = __shfl_sync(0xffffffff, threadIdx.x / WARP_SIZE, 0);

    if constexpr (UseGroups)
    {
        if (warpIdx >= numGroup)
        {
            return;
        }
    }

    // note that for invalid scores, we simply use a negative value:
    // they work well even with the compacted format used in topK, and
    // sigmoid / bias activated scores cannot be negative
    static constexpr float invalidScoreFloat = -1.F;
    const OutputT invalidScore = OutputT{invalidScoreFloat};

    // load bias already; each warp represents one expert group
    auto threadExpert = threadIdx.x;
    bool expertSelected = threadExpert < numExperts;
    if constexpr (UseGroups)
    {
        threadExpert = warpIdx * numExpertsPerGroup + laneIdx;
        expertSelected = laneIdx < numExpertsPerGroup;
    }

    auto scoreIdx = int64_t{blockIdx.x} * int64_t{numExperts} + threadExpert;
    auto biasVal = expertSelected ? static_cast<float>(routingBias[threadExpert]) : invalidScoreFloat;
    topkValues += blockIdx.x * topk;
    topkIndices += blockIdx.x * topk;

    // get our assigned thread score; each warp represents one expert group
    float score = expertSelected ? static_cast<float>(scores[scoreIdx]) : invalidScoreFloat;
    auto scoreSigmoid = sigmoid_accurate(score);
    // write the sigmoid score to shared for later use
    if (expertSelected)
    {
        smemScoreSigmoid[threadExpert] = scoreSigmoid;
    }

    // get the score with bias
    // note that with invalid values, because sigmoid is < 1 and bias is -1,
    // we must get a negative value, which is smaller than any valid value
    auto scoreBias = float{scoreSigmoid + float{biasVal}};

    if (expertSelected)
    {
        smemScoreBias[threadExpert] = scoreBias;
    }

    // registers for top group score reduction
    float topExpGroupScores[NumTopGroupScores];
    [[maybe_unused]] int32_t topExpGroupIdx[NumTopGroupScores];
    float topGroups[MaxNumTopGroups]; // bound of numGroup
    int32_t topGroupIdx[MaxNumTopGroups];
    float expertScoreGroup[MaxNumTopGroups];
    int32_t expertIdxGroup[MaxNumTopGroups];
    float topScores[MaxNumTopExperts]; // bound of topk
    int32_t topExperts[MaxNumTopExperts];

    if constexpr (UseGroups)
    {
        reduce_topk::reduceTopK(warp, topExpGroupScores, topExpGroupIdx, scoreBias, threadExpert,
            /* minValue */ invalidScoreFloat);

        // get the final group score and write it to shared
        if (laneIdx == 0)
        {
            auto groupScore = topExpGroupScores[0] + topExpGroupScores[1];
            smemGroupScores[warpIdx] = groupScore;
        }
    }

    // make group scores available to all warps
    __syncthreads();

    if constexpr (UseGroups)
    {
        if (warpIdx == 0)
        {
            // a single warp performs the selection of top groups, and goes on to select the final experts
            float groupScore = laneIdx < numGroup ? smemGroupScores[laneIdx] : invalidScoreFloat;

            reduce_topk::reduceTopK(warp, topGroups, topGroupIdx, groupScore, laneIdx,
                /* minValue */ invalidScoreFloat);

            // final expert selection: get relevant indexes and scores from shared

#pragma unroll
            for (int ii = 0; ii < MaxNumTopGroups; ++ii)
            { // bound of numGroup
                auto groupIdx = topGroupIdx[ii];
                expertIdxGroup[ii] = groupIdx * numExpertsPerGroup + laneIdx;

                expertScoreGroup[ii]
                    = groupIdx < numGroup && expertSelected ? smemScoreBias[expertIdxGroup[ii]] : invalidScoreFloat;
            }

            tensorrt_llm::kernels::reduce_topk::reduceTopK(warp, topScores, topExperts, expertScoreGroup,
                expertIdxGroup,
                /* minValue */ invalidScoreFloat, topk);
        }
    }
    else if constexpr (MaxNumExperts > MaxNumExpertsUnit)
    {
        // without groups, and the expert number is larger than MaxNumExpertsUnit,
        // we need to use multiple warps to calculate the intermediate topk results

        int constexpr NumExpertWarps = (MaxNumExperts - 1) / MaxNumExpertsUnit + 1;
        int constexpr NumInterTopK = NumExpertWarps * MaxNumTopExperts;
        __shared__ float __attribute((aligned(128))) smemInterTopScores[NumInterTopK];
        __shared__ int32_t __attribute((aligned(128))) smemInterTopExperts[NumInterTopK];
        if (warpIdx < NumExpertWarps)
        {
            int offset = warpIdx * WARP_SIZE * MaxNumTopGroups;
#pragma unroll
            for (int ii = 0; ii < MaxNumTopGroups; ++ii)
            {
                auto expertIdx = ii * WARP_SIZE + laneIdx;
                expertIdxGroup[ii] = offset + expertIdx;
                expertScoreGroup[ii]
                    = offset + expertIdx < numExperts ? smemScoreBias[offset + expertIdx] : invalidScoreFloat;
            }
            reduce_topk::reduceTopK(warp, topScores, topExperts, expertScoreGroup, expertIdxGroup,
                /* minValue */ invalidScoreFloat, topk);

            if (laneIdx < topk)
            {
                smemInterTopScores[warpIdx * MaxNumTopExperts + laneIdx] = topScores[laneIdx];
                smemInterTopExperts[warpIdx * MaxNumTopExperts + laneIdx] = topExperts[laneIdx];
            }
        }
        __syncthreads();
        if (warpIdx == 0)
        {
            int constexpr NumInterTopKPerThread = (NumInterTopK * NumExpertWarps - 1) / WARP_SIZE + 1;
            float intermidiateScore[NumInterTopKPerThread];
            int32_t intermidiateExpert[NumInterTopKPerThread];
            for (int i = laneIdx; i < NumInterTopKPerThread * WARP_SIZE; i += WARP_SIZE)
            {
                int ii = i / WARP_SIZE;
                if (i < NumInterTopK)
                {
                    intermidiateScore[ii] = smemInterTopScores[i];
                    intermidiateExpert[ii] = smemInterTopExperts[i];
                }
                else
                {
                    intermidiateScore[ii] = invalidScoreFloat;
                    intermidiateExpert[ii] = MaxNumExperts - 1;
                }
            }
            reduce_topk::reduceTopK(warp, topScores, topExperts, intermidiateScore, intermidiateExpert,
                /* minValue */ invalidScoreFloat, topk);
        }
    }
    else
    {
        // without groups, and the expert number is smaller than MaxNumExpertsUnit
        // each thread just takes `MaxNumTopGroups` experts
        if (warpIdx == 0)
        {
#pragma unroll
            for (int ii = 0; ii < MaxNumTopGroups; ++ii)
            {
                auto expertIdx = ii * WARP_SIZE + laneIdx;
                expertIdxGroup[ii] = expertIdx;
                expertScoreGroup[ii] = expertIdx < numExperts ? smemScoreBias[expertIdx] : invalidScoreFloat;
            }
            reduce_topk::reduceTopK(warp, topScores, topExperts, expertScoreGroup, expertIdxGroup,
                /* minValue */ invalidScoreFloat, topk);
        }
    }

    if (warpIdx == 0)
    {
        // determine our lane's expert index and write to output
        int32_t expertIdx = laneIdx < topk ? topExperts[laneIdx] : MaxNumExperts - 1;
        // norm the value
        float scoreNorm = laneIdx < topk ? smemScoreSigmoid[expertIdx] : 0.F;
        auto redNorm = cg::reduce(warp, scoreNorm, cg::plus<float>{});
        auto finalScore = static_cast<OutputT>(scoreNorm * routedScalingFactor / (redNorm + 1e-20));
        // store the topk scores and experts to output
        if (laneIdx < topk)
        {
            topkValues[laneIdx] = static_cast<OutputT>(finalScore);
            topkIndices[laneIdx] = expertIdx;
        }
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <typename InputT, typename BiasT, typename OutputT, typename IdxT>
void invokeNoAuxTc(InputT* scores, BiasT* bias, OutputT* topk_values, IdxT* topk_indices, int64_t const num_tokens,
    int64_t const num_experts, int64_t const n_group, int64_t const topk_group, int64_t const topk,
    double const routed_scaling_factor, hipStream_t const stream)
{

    // Check if we can use the optimized deepseek_v3_topk_kernel
    bool const is_single_group = (n_group == 1) && (num_experts <= NumKimiK2Experts);

    int64_t const experts_per_group = num_experts / n_group;
    bool const is_multi_group = (n_group != 1) && (num_experts <= NumDeepseekExperts)
        && (experts_per_group <= WARP_SIZE) && (experts_per_group * topk_group <= MaxNumExpertsUnit);

    if (is_single_group || is_multi_group)
    {
        hipLaunchConfig_t config;
        auto* kernel_instance = &deepseek_v3_topk_kernel<InputT, BiasT, OutputT, IdxT, NumDeepseekExperts, true>;
        int num_threads = NumDeepseekExperts;
        if (is_single_group)
        {
            if (num_experts > MaxNumExpertsUnit)
            {
                kernel_instance = &deepseek_v3_topk_kernel<InputT, BiasT, OutputT, IdxT, NumKimiK2Experts, false>;
                num_threads = NumKimiK2Experts;
            }
            else
            {
                kernel_instance = &deepseek_v3_topk_kernel<InputT, BiasT, OutputT, IdxT, MaxNumExpertsUnit, false>;
                num_threads = MaxNumExpertsUnit;
            }
        }

        config.gridDim = num_tokens;
        config.blockDim = num_threads;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attrs[1];
        attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.numAttrs = 1;
        config.attrs = attrs;

        cudaLaunchKernelEx(&config, kernel_instance, scores, topk_values, topk_indices, bias, num_tokens, n_group,
            topk_group, topk, num_experts, num_experts / n_group, routed_scaling_factor);
        sync_check_cuda_error(stream);
    }
    else
    {
        // TODO: call the generic path (previous implementation) or signal unsupported config.
        TLLM_CHECK_WITH_INFO(false,
            "invokeNoAuxTc: unsupported configuration (n_group=%ld, num_experts=%ld, topk_group=%ld). Please use "
            "original pytorch implementation.",
            n_group, num_experts, topk_group);
    }
}

#define INSTANTIATE_NOAUX_TC(InputT, BiasT, OutputT, IdxT)                                                             \
    template void invokeNoAuxTc<InputT, BiasT, OutputT, IdxT>(InputT * scores, BiasT * bias, OutputT * topk_values,    \
        IdxT * topk_indices, int64_t const num_tokens, int64_t const num_experts, int64_t const n_group,               \
        int64_t const topk_group, int64_t const topk, double const routed_scaling_factor, hipStream_t const stream);

INSTANTIATE_NOAUX_TC(float, float, float, int32_t);
INSTANTIATE_NOAUX_TC(float, half, float, int32_t);

INSTANTIATE_NOAUX_TC(half, float, half, int32_t);
INSTANTIATE_NOAUX_TC(half, half, half, int32_t);

#ifdef ENABLE_BF16
INSTANTIATE_NOAUX_TC(float, __hip_bfloat16, float, int32_t);
INSTANTIATE_NOAUX_TC(half, __hip_bfloat16, half, int32_t);

INSTANTIATE_NOAUX_TC(__hip_bfloat16, __hip_bfloat16, __hip_bfloat16, int32_t);
INSTANTIATE_NOAUX_TC(__hip_bfloat16, float, __hip_bfloat16, int32_t);
INSTANTIATE_NOAUX_TC(__hip_bfloat16, half, __hip_bfloat16, int32_t);
#endif

} // namespace tensorrt_llm::kernels
