#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/speculativeDecoding/externalDraftTokensKernels.h"
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::kernels::speculative_decoding
{
namespace
{

template <typename T>
__global__ void maskTargetLogitsKernel(T* targetLogits, SizeType32 const* batchSlots, SizeType32 beamWidth,
    SizeType32 vocabSize, FinishedState const* finishedInput, SizeType32 maxBatchSize,
    SizeType32* outputIdsAfterSampling, SizeType32* runtimeTopKDevicePtr, bool* maskBuffer)
{
    /**
     * @brief Masking the selected token to -inf as was done in Huggingface TopK/TopP Logits Warper
     * https://github.com/huggingface/transformers/blob/2e24ee4dfa39cc0bc264b89edbccc373c8337086/src/transformers/generation/logits_process.py#L533
     */

    auto const bid = blockIdx.x;
    auto const batchIdx = bid / beamWidth;
    auto const tid = static_cast<SizeType32>(threadIdx.x);
    auto const batchSlot = batchSlots[batchIdx];

    constexpr bool IS_HALF = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_HALF) ? HALF_FLT_MAX : FLT_MAX;

    auto targetLogitsBatch = targetLogits + batchIdx * vocabSize;
    auto& finishedState = finishedInput[batchSlot];

    auto* outputIdsAfterSamplingPtr = outputIdsAfterSampling + batchSlot * vocabSize;
    auto* maskBufferBatch = maskBuffer + batchSlot * vocabSize;

    if (finishedState.isSkipDecoding() || finishedState.isFinished())
    {
        return;
    }

    __shared__ SizeType32 tokensToMask;

    if (tid == 0)
    {
        tokensToMask = runtimeTopKDevicePtr[batchSlot];
    }
    __syncthreads();

    for (SizeType32 vIdx = tid; vIdx < vocabSize; vIdx += static_cast<SizeType32>(blockDim.x))
    {
        if (outputIdsAfterSamplingPtr[vIdx] == -1)
        { // we need to find the -1 boundary from returnAllTopP outputIds if topK == 0 or number of topP indices < topK
            tokensToMask = vIdx;
        }
        maskBufferBatch[vIdx] = false;
    }

    __syncthreads();
    if (tid == 0 && tokensToMask == 0)
    {
        // all tokens are selected if topK == 0 && topP ~= 1.0f
        // in this case tokensToMask = vocabSize
        tokensToMask = vocabSize;
    }
    __syncthreads();

    for (SizeType32 vIdx = tid; vIdx < tokensToMask; vIdx += static_cast<SizeType32>(blockDim.x))
    {
        auto tokenToMask = outputIdsAfterSamplingPtr[vIdx];
        maskBufferBatch[tokenToMask] = true;
    }

    __syncthreads();

    for (SizeType32 vIdx = tid; vIdx < vocabSize; vIdx += static_cast<SizeType32>(blockDim.x))
    {
        if (!maskBufferBatch[vIdx])
        {
            targetLogitsBatch[vIdx] = -MAX_T_VAL;
        }
    }
}

template <typename T>
__global__ void acceptDraftTokensKernel(T const* draftProbs, T* targetProbs, SizeType32 const* numsDraftTokens,
    bool const* batchUseDraftLogits, TokenIdType const* draftIds, FinishedState const* finishedInput,
    FinishedState* finishedOutput, hiprandState_t* hiprandState, SizeType32 const* batchSlots, SizeType32 maxDraftTokens,
    SizeType32 beamWidth, SizeType32 vocabSize, bool randomThreshold, float constantThreshold, SizeType32 step,
    bool* batchIsAccepted, SizeType32* targetOutputIds)
{
    auto const bid = blockIdx.x;
    auto const draftTokenIdx = step;
    auto const batchIdx = bid / beamWidth;
    auto const beamIdx = bid % beamWidth;
    auto const batchSlot = batchSlots[batchIdx];
    auto const batchSlotBeamWidth = batchSlot * beamWidth + beamIdx;
    auto const tid = static_cast<SizeType32>(threadIdx.x);

    auto const numDraftTokens = numsDraftTokens[batchSlotBeamWidth];
    auto const useDraftLogits = batchUseDraftLogits[batchSlotBeamWidth];

    if (numDraftTokens == 0 || draftTokenIdx > numDraftTokens || finishedInput[batchSlot].isSkipDecoding()
        || finishedInput[batchSlot].isFinished())
    {
        if (tid == 0)
        {
            batchIsAccepted[batchSlot] = true;

            // either finished or skip decode in previous step, this step don't need decoding
            finishedOutput[batchSlot].setSkipDecoding();

            // if previous step is finished, write the state to next step too
            if (finishedInput[batchSlot].isFinished())
            {
                finishedOutput[batchSlot] = finishedInput[batchSlot];
            }
        }
        return;
    }

    if (draftTokenIdx == numDraftTokens)
    {
        if (tid == 0)
        {
            batchIsAccepted[batchSlot] = false;
            finishedOutput[batchSlot].setSkipDecoding();
        }
        return;
    }
    // else (draftTokenIdx < numDraftTokens)

    auto const logitsOffset = (batchSlot * maxDraftTokens + draftTokenIdx) * beamWidth * vocabSize;
    auto const draftProbsBatch = draftProbs + logitsOffset;
    auto const targetProbsBatch = targetProbs + (batchIdx * beamWidth * vocabSize);

    __shared__ bool isAccepted;
    __shared__ T sSumVal;
    if (tid == 0)
    {
        auto const draftOutputTokenId = draftIds[batchSlot * maxDraftTokens + draftTokenIdx];
        if (useDraftLogits)
        {
            float threshold = randomThreshold ? hiprand_uniform(hiprandState + batchSlot) : constantThreshold;
            auto const targetProb = static_cast<float>(targetProbsBatch[draftOutputTokenId]);
            auto const draftProb = static_cast<float>(draftProbsBatch[draftOutputTokenId]);
            isAccepted = targetProb > threshold * draftProb;
        }
        else
        {
            // Check if draft tokens are the same as target tokens
            isAccepted = targetOutputIds[batchSlot] == draftOutputTokenId;
        }
        if (!isAccepted)
        {
            finishedOutput[batchSlot].setSkipDecoding();
        }
        batchIsAccepted[batchSlot] = isAccepted;
    }

    __syncthreads();

    if (useDraftLogits && !isAccepted)
    {
        // correct target distribution
        T const zeroVal = static_cast<T>(0.0F);
        T sumVal = zeroVal;
        for (SizeType32 vIdx = tid; vIdx < vocabSize; vIdx += static_cast<SizeType32>(blockDim.x))
        {
            targetProbsBatch[vIdx] -= draftProbsBatch[vIdx];
            targetProbsBatch[vIdx] = targetProbsBatch[vIdx] >= zeroVal ? targetProbsBatch[vIdx] : zeroVal;
            sumVal += targetProbsBatch[vIdx];
        }
        sumVal = blockReduceSum<T>(sumVal);
        if (tid == 0)
        {
            sSumVal = sumVal;
        }
        __syncthreads();

        for (SizeType32 vIdx = tid; vIdx < vocabSize; vIdx += static_cast<SizeType32>(blockDim.x))
        {
            targetProbsBatch[vIdx] /= sSumVal;
        }
    }
}

__global__ void forwardAcceptedTokensKernel(SizeType32 batchSize, SizeType32 const* batchSlots, bool* batchIsAccepted,
    SizeType32* sequenceLengths, TokenIdType const* draftIds, TokenIdType** idsPtrs, SizeType32 step,
    SizeType32 maxDraftTokens, TokenIdType const* endIds, FinishedState* finishedOutput)
{
    auto index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
    for (SizeType32 bi = index; bi < batchSize; bi += static_cast<SizeType32>(gridDim.x * blockDim.x))
    {
        auto const batchSlot = batchSlots[bi];
        if (batchIsAccepted[batchSlot] && !finishedOutput[batchSlot].isSkipDecoding()
            && !finishedOutput[batchSlot].isFinished())
        {
            auto const curSeqLen = sequenceLengths[batchSlot];
            auto const draftTokenIdx = step;
            auto const draftOutputTokenId = draftIds[batchSlot * maxDraftTokens + draftTokenIdx];
            auto* outputIdsRequestPtr = idsPtrs[batchSlot];
            auto const outIdx = curSeqLen;
            outputIdsRequestPtr[outIdx] = draftOutputTokenId;
            if (outputIdsRequestPtr[outIdx] == endIds[batchSlot])
            {
                finishedOutput[batchSlot].setFinishedEOS();
                // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be
                // outputted
            }
            else
            {
                // We don't need to set output finished state as it is assumed to be in non finished state
                sequenceLengths[batchSlot] += 1;
            }
        }
    }
} // namespace

} // namespace

template <typename T>
void invokeMaskTargetLogits(SizeType32 batchSize, T* targetLogits, SizeType32 const* batchSlots, SizeType32 beamWidth,
    SizeType32 vocabSizePadded, FinishedState const* finishedInput, SizeType32 maxBatchSize,
    SizeType32* outputIdsAfterSampling, SizeType32* runtimeTopKDevicePtr, bool* maskBuffer, hipStream_t stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    TLLM_CHECK(beamWidth == 1);
    {
        dim3 block(1024);
        dim3 grid(batchSize * beamWidth);
        maskTargetLogitsKernel<<<grid, block, 0, stream>>>(targetLogits, batchSlots, beamWidth, vocabSizePadded,
            finishedInput, maxBatchSize, outputIdsAfterSampling, runtimeTopKDevicePtr, maskBuffer);
    }
    sync_check_cuda_error(stream);
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void invokeAcceptDraftTokens(SizeType32 batchSize, T* draftProbs, T* targetProbs, SizeType32 const* numsDraftTokens,
    bool const* batchUseDraftLogits, TokenIdType const* draftIds, FinishedState const* finishedInput,
    FinishedState* finishedOutput, hiprandState_t* hiprandState, SizeType32 const* batchSlots, SizeType32 maxDraftTokens,
    SizeType32 beamWidth, SizeType32 vocabSizePadded, bool randomThreshold, float constantThreshold, SizeType32 step,
    bool* batchIsAccepted, SizeType32* targetOutputIds, hipStream_t stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    TLLM_CHECK(beamWidth == 1);
    {
        dim3 block(1024);
        dim3 grid(batchSize * beamWidth);
        acceptDraftTokensKernel<<<grid, block, 0, stream>>>(draftProbs, targetProbs, numsDraftTokens,
            batchUseDraftLogits, draftIds, finishedInput, finishedOutput, hiprandState, batchSlots, maxDraftTokens,
            beamWidth, vocabSizePadded, randomThreshold, constantThreshold, step, batchIsAccepted, targetOutputIds);
    }
    sync_check_cuda_error(stream);
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template void invokeMaskTargetLogits(SizeType32 batchSize, float* targetLogits, SizeType32 const* batchSlots,
    SizeType32 beamWidth, SizeType32 vocabSizePadded, FinishedState const* finishedInput, SizeType32 maxBatchSize,
    SizeType32* outputIdsAfterSampling, SizeType32* runtimeTopKDevicePtr, bool* maskBuffer, hipStream_t stream);
template void invokeMaskTargetLogits(SizeType32 batchSize, half* targetLogits, SizeType32 const* batchSlots,
    SizeType32 beamWidth, SizeType32 vocabSizePadded, FinishedState const* finishedInput, SizeType32 maxBatchSize,
    SizeType32* outputIdsAfterSampling, SizeType32* runtimeTopKDevicePtr, bool* maskBuffer, hipStream_t stream);

template void invokeAcceptDraftTokens(SizeType32 batchSize, float* draftProbs, float* targetProbs,
    SizeType32 const* numsDraftTokens, bool const* batchUseDraftLogits, TokenIdType const* draftIds,
    FinishedState const* finishedInput, FinishedState* finishedOutput, hiprandState_t* hiprandState,
    SizeType32 const* batchSlots, SizeType32 maxDraftTokens, SizeType32 beamWidth, SizeType32 vocabSizePadded,
    bool randomThreshold, float constantThreshold, SizeType32 step, bool* batchIsAccepted, SizeType32* targetOutputIds,
    hipStream_t stream);
template void invokeAcceptDraftTokens(SizeType32 batchSize, half* draftProbs, half* targetProbs,
    SizeType32 const* numsDraftTokens, bool const* batchUseDraftLogits, TokenIdType const* draftIds,
    FinishedState const* finishedInput, FinishedState* finishedOutput, hiprandState_t* hiprandState,
    SizeType32 const* batchSlots, SizeType32 maxDraftTokens, SizeType32 beamWidth, SizeType32 vocabSizePadded,
    bool randomThreshold, float constantThreshold, SizeType32 step, bool* batchIsAccepted, SizeType32* targetOutputIds,
    hipStream_t stream);

void invokeForwardAcceptedTokens(SizeType32 batchSize, SizeType32 const* batchSlots, bool* batchIsAccepted,
    SizeType32* outputSequenceLengths, TokenIdType const* draftIds, TokenIdType** idsPtrs, SizeType32 step,
    SizeType32 maxDraftTokens, TokenIdType const* endIds, FinishedState* finishedOutput, hipStream_t stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    dim3 block(std::min(static_cast<uint32_t>(batchSize), 256u));
    dim3 grid(divUp(static_cast<uint32_t>(batchSize), block.x));
    forwardAcceptedTokensKernel<<<grid, block, 0, stream>>>(batchSize, batchSlots, batchIsAccepted,
        outputSequenceLengths, draftIds, idsPtrs, step, maxDraftTokens, endIds, finishedOutput);
    sync_check_cuda_error(stream);
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}
} // namespace tensorrt_llm::kernels::speculative_decoding
