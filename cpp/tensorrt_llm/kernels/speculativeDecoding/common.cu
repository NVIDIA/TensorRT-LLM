#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/common/workspace.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include "tensorrt_llm/kernels/speculativeDecoding/common.h"
#include "tensorrt_llm/runtime/runtimeKernels.h"

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::kernels::speculative_decoding
{
template <int32_t BLOCK_SIZE>
__global__ void packAcceptedPaths(SizeType32* acceptedLengthsCumSum, SizeType32* pathsOffsets,
    SizeType32 const* acceptedLengths, SizeType32 const* bestPathIds, SizeType32 const* paths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 engineBatchSize, SizeType32 numPaths,
    SizeType32 maxPathLen, bool isPathsLinearBatchIdx)
{
    // Specialize BlockScan for a 1D block of 128 threads of type int
    typedef hipcub::BlockScan<SizeType32, BLOCK_SIZE> BlockScan;

    // Allocate shared memory for BlockScan
    __shared__ typename BlockScan::TempStorage tempStorage;
    auto const batchSizeRounded = ((engineBatchSize + BLOCK_SIZE - 1) / BLOCK_SIZE) * BLOCK_SIZE;
    __shared__ SizeType32 currentCumSum;
    __shared__ SizeType32 currentValidIdx;
    if (threadIdx.x == 0)
    {
        currentCumSum = 0;
        currentValidIdx = 0;
    }

    __syncthreads();

    for (auto bi = static_cast<SizeType32>(threadIdx.x); bi < batchSizeRounded;
         bi += static_cast<SizeType32>(blockDim.x))
    {
        auto valid = bi < engineBatchSize;
        auto const batchSlot = valid ? batchSlots[bi] : 0;
        if (batchSlot < 0)
        {
            valid = false;
        }
        auto const acceptedLen = valid ? acceptedLengths[batchSlot] - 1 : 0;
        SizeType32 cumSum;
        BlockScan(tempStorage).ExclusiveSum(acceptedLen + currentCumSum, cumSum);
        __syncthreads();
        SizeType32 validIndex;
        BlockScan(tempStorage).ExclusiveSum(static_cast<SizeType32>(valid) + currentValidIdx, validIndex);
        if (threadIdx.x == blockDim.x - 1)
        {
            currentCumSum = cumSum;
            currentValidIdx = validIndex;
        }
        __syncthreads();

        if (valid)
        {
            acceptedLengthsCumSum[validIndex] = cumSum;
            auto const pathBatchIdx = isPathsLinearBatchIdx ? bi : batchSlot;
            auto const bestPathIdx = bestPathIds[pathBatchIdx];
            auto const pathIdx = flat_index3(pathBatchIdx, bestPathIdx, 0, numPaths, maxPathLen);
            for (SizeType32 ti = 0; ti < acceptedLen; ++ti)
            {
                pathsOffsets[cumSum + ti] = paths[pathIdx + ti + 1] - 1;
            }
        }
    }
    if (threadIdx.x == 0)
    {
        acceptedLengthsCumSum[batchSize] = currentCumSum;
    }
}

void invokePackAcceptedPaths(SizeType32* acceptedLengthsCumSum, SizeType32* pathsOffsets,
    SizeType32 const* acceptedLengths, SizeType32 const* bestPathIds, SizeType32 const* paths,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 engineBatchSize, SizeType32 numPaths,
    SizeType32 maxPathLen, bool isPathsLinearBatchIdx, hipStream_t stream)
{
    constexpr SizeType32 BLOCK_SIZE = 1024;
    packAcceptedPaths<BLOCK_SIZE><<<1, BLOCK_SIZE, 0, stream>>>(acceptedLengthsCumSum, pathsOffsets, acceptedLengths,
        bestPathIds, paths, batchSlots, batchSize, engineBatchSize, numPaths, maxPathLen, isPathsLinearBatchIdx);
}

namespace
{
__device__ __forceinline__ int4 reduceMaxInt4(int4 const& a, int4 const& b)
{
    return a.x >= b.x ? a : b;
}

template <typename T, SizeType32 BLOCK_SIZE>
__global__ void acceptDraftTokensByIdsWithPaths(TokenIdType* outputIds, TokenIdType const* draftIds,
    TokenIdType const* targetIds, SizeType32* sequenceLengths, SizeType32* acceptedLengths,
    FinishedState* finishedFinal, SizeType32 const* batchSlots, SizeType32 const* paths, TokenIdType const* endIds,
    T const** medusaLogits, T const** logitsPtrs, SizeType32* curTokensPerStep, SizeType32 const* targetTokensPerStep,
    SizeType32* bestPathIds, SizeType32 batchSize, SizeType32 vocabSize, SizeType32 maxBatchSize, SizeType32 maxSeqLen,
    SizeType32 maxDraftPathLen, SizeType32 maxDecodingTokens)
{
    auto const batchIdx = static_cast<SizeType32>(blockIdx.x);
    auto const batchSlot = batchSlots == nullptr ? batchIdx : batchSlots[batchIdx];
    auto const inputLength = sequenceLengths == nullptr ? 0 : sequenceLengths[batchSlot];
    auto const endId = endIds == nullptr ? -1 : endIds[batchSlot];
    auto const numTokensPerStep = curTokensPerStep == nullptr ? maxDecodingTokens : curTokensPerStep[batchSlot];
    auto const maxPathLen = maxDraftPathLen + 1;

    int4 partialMax{-1, -1, 0, 0};
    // Go over different paths and construct implicit sequences
    for (auto pathIdx = static_cast<SizeType32>(threadIdx.x); pathIdx < maxDecodingTokens;
         pathIdx += static_cast<SizeType32>(blockDim.x))
    {
        auto acceptedLength = maxPathLen;
        auto const pathOffset = flat_index3(batchSlot, pathIdx, 0, maxDecodingTokens, maxPathLen);
        bool hasEnd = false;

        auto const tokenId = paths[pathOffset];
        // Continue if path does not exist
        if (tokenId == -1)
        {
            continue;
        }
        auto const targetTokenIdx = batchSlot * maxDecodingTokens + tokenId;
        auto targetToken = targetIds[targetTokenIdx];
        auto nextIdx = tokenId;

        // Go along the path
        for (SizeType32 ti = 1; ti < maxPathLen; ++ti)
        {
            auto const tokenId = paths[pathOffset + ti];
            // Break if path terminates
            if (tokenId == -1)
            {
                hasEnd = endIds == nullptr ? false
                                           : targetToken == endId; // check if last token is EOS when path terminates.
                acceptedLength = hasEnd ? ti - 1 : ti;
                break;
            }
            auto const targetTokenIdx = batchSlot * maxDecodingTokens + tokenId;
            auto const draftTokenIdx = batchSlot * (maxDecodingTokens - 1) + tokenId - 1;
            // In context phase, no draft tokens are given. Set draft token to -1 to get guaranteed rejection
            auto const draftToken = tokenId >= numTokensPerStep ? -1 : draftIds[draftTokenIdx];
            // Check if draft tokens are the same as target tokens
            bool const accepted = draftToken == targetToken;
            hasEnd = endIds == nullptr ? false : targetToken == endId;
            if (!accepted || hasEnd)
            {
                acceptedLength = hasEnd ? ti - 1 : ti;
                break;
            }
            targetToken = targetIds[targetTokenIdx];
            nextIdx = tokenId;
        }
        // Get longest path of the thread
        if (partialMax.x < acceptedLength)
        {
            partialMax.x = acceptedLength;
            partialMax.y = pathIdx;
            partialMax.z = hasEnd;
            partialMax.w = nextIdx;
        }
    }

    // Get the longest path of the block (request)
    typedef hipcub::BlockReduce<int4, BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage tempStorage;
    int4 total = BlockReduce(tempStorage).Reduce(partialMax, reduceMaxInt4);

    __shared__ int4 totalShared;
    if (threadIdx.x == 0)
    {
        totalShared = total;
    }

    __syncthreads();

    auto const acceptedLength = totalShared.x;
    auto const bestPathIdx = totalShared.y;
    auto const bestNextIdx = numTokensPerStep == 1 ? 0 : totalShared.w;
    auto const pathOffset = flat_index3(batchSlot, bestPathIdx, 0, maxDecodingTokens, maxPathLen);
    for (auto ti = static_cast<SizeType32>(threadIdx.x); ti < acceptedLength; ti += static_cast<SizeType32>(blockDim.x))
    {
        auto const tokenId = paths[pathOffset + ti];
        auto const targetSrcTokenIdx = batchSlot * maxDecodingTokens + tokenId;
        auto const outputTokenIdx = batchSlot * maxSeqLen + inputLength + ti;
        auto const targetToken = targetIds[targetSrcTokenIdx];
        // Copy accepted tokens to the sequence with draft tokens (outputIds === outputIds)
        outputIds[outputTokenIdx] = targetToken;
    }

    // Leading thread reconstructs winning path and sets new data
    if (threadIdx.x == 0)
    {
        auto const hasEnd = totalShared.z;
        // Set end condition
        if (hasEnd && finishedFinal)
        {
            finishedFinal[batchSlot].setFinishedEOS();
        }
        // Make correction to the sequence length
        if (sequenceLengths)
        {
            sequenceLengths[batchSlot] += acceptedLength;
        }
        acceptedLengths[batchSlot] = acceptedLength;
        // In Medusa decoding step, number of draft tokens is 0 and must be updated for the next steps
        if (curTokensPerStep && targetTokensPerStep && numTokensPerStep == 1)
        {
            curTokensPerStep[batchSlot] = targetTokensPerStep[batchSlot];
        }
        bestPathIds[batchSlot] = bestPathIdx;
    }

    // Prepare logits pointers to respective logits from Medusa Heads for the all-top-K sampling kernel
    if (medusaLogits && logitsPtrs)
    {
        for (auto hi = static_cast<SizeType32>(threadIdx.x); hi < maxDraftPathLen;
             hi += static_cast<SizeType32>(blockDim.x))
        {
            logitsPtrs[batchIdx * maxDraftPathLen + hi]
                = medusaLogits[batchSlot * maxDraftPathLen + hi] + flat_index2(bestNextIdx, 0, vocabSize);
        }
    }
}
} // namespace

template <typename T>
void acceptDraftTokensByIdsWithPaths(AcceptDraftTokensByIdsWithPathsParams<T> const& params)
{
    SizeType32 constexpr BLOCK_SIZE = 256;
    dim3 block(BLOCK_SIZE);
    dim3 grid(params.batchSize);
    acceptDraftTokensByIdsWithPaths<T, BLOCK_SIZE><<<grid, block, 0, params.stream>>>(params.outputIds, params.draftIds,
        params.targetIds, params.sequenceLengths, params.acceptedLengths, params.finishedFinal, params.batchSlots,
        params.paths, params.endIds, params.medusaLogits, params.logitsPtrs, params.curTokensPerStep,
        params.targetTokensPerStep, params.bestPathIds, params.batchSize, params.vocabSize, params.maxBatchSize,
        params.maxSeqLen, params.maxDraftPathLen, params.maxDecodingTokens);
}

template void acceptDraftTokensByIdsWithPaths(AcceptDraftTokensByIdsWithPathsParams<float> const& params);
template void acceptDraftTokensByIdsWithPaths(AcceptDraftTokensByIdsWithPathsParams<__half> const& params);

namespace
{
template <typename T>
__global__ void maskLogitsBasedOnEntropyKernel(T** logitsPtrs, TokenIdType** outputIdsPtrs, bool* skipDecode,
    TokenIdType* outputIds, float* runtimeMultinomialTopP, T const* probsPtr, float const* entropies,
    SizeType32 const* generationLengths, float const* posteriorThresholds, float const* posteriorAlphas,
    float const* temperatures, SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 maxDecodingTokens,
    SizeType32 vocabSize)
{
    auto const tix = blockIdx.z * blockDim.x + threadIdx.x;
    auto const batchIdx = static_cast<SizeType32>(blockIdx.x);
    auto const tokenIdx = static_cast<SizeType32>(blockIdx.y);
    auto const vocabIdx = static_cast<SizeType32>(tix);

    auto const batchSlot = batchSlots == nullptr ? batchIdx : batchSlots[batchIdx];
    auto const valid = tokenIdx < generationLengths[batchSlot] && vocabIdx < vocabSize;

    if (vocabIdx == 0)
    {
        skipDecode[batchIdx * maxDecodingTokens + tokenIdx] = !valid;
        outputIdsPtrs[batchIdx * maxDecodingTokens + tokenIdx] = outputIds + batchIdx * maxDecodingTokens + tokenIdx;
        if (temperatures[batchSlot] < 1e-6f)
        {
            // Greedy sampling with temp = 0.
            runtimeMultinomialTopP[batchSlot * maxDecodingTokens + tokenIdx] = 0.f;
        }
        else
        {
            runtimeMultinomialTopP[batchSlot * maxDecodingTokens + tokenIdx] = 1.f;
        }
    }

    if (!valid)
    {
        return;
    }

    auto const posteriorThreshold = posteriorThresholds[batchSlot];
    auto const posteriorAlpha = posteriorAlphas[batchSlot];
    auto const entropy = entropies[batchSlot * maxDecodingTokens + tokenIdx];

    auto const prob = static_cast<float>(probsPtr[(batchSlot * maxDecodingTokens + tokenIdx) * vocabSize + vocabIdx]);
    auto const threshold = min(posteriorThreshold, posteriorAlpha * expf(-entropy));
    if (prob < threshold)
    {
        logitsPtrs[batchIdx * maxDecodingTokens + tokenIdx][vocabIdx] = -FLT_MAX;
    }
}
} // namespace

template <typename T>
void maskLogitsBasedOnEntropy(T** logitsPtrs, TokenIdType** outputIdsPtrs, bool* skipDecode, TokenIdType* outputIds,
    float* runtimeMultinomialTopP, T const* probsPtr, float const* entropies, SizeType32 const* generationLengths,
    float const* posteriorThresholds, float const* posteriorAlphas, float const* temperatures,
    SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 maxDecodingTokens, SizeType32 vocabSize,
    hipStream_t stream)
{
    SizeType32 constexpr BLOCK_SIZE = 512;
    SizeType32 numBlocks = divUp(vocabSize, BLOCK_SIZE);
    dim3 grid(batchSize, maxDecodingTokens, numBlocks);
    maskLogitsBasedOnEntropyKernel<<<grid, BLOCK_SIZE, 0, stream>>>(logitsPtrs, outputIdsPtrs, skipDecode, outputIds,
        runtimeMultinomialTopP, probsPtr, entropies, generationLengths, posteriorThresholds, posteriorAlphas,
        temperatures, batchSlots, batchSize, maxDecodingTokens, vocabSize);
}

template void maskLogitsBasedOnEntropy(float** logitsPtrs, TokenIdType** outputIdsPtrs, bool* skipDecode,
    TokenIdType* outputIds, float* runtimeMultinomialTopP, float const* probsPtr, float const* entropies,
    SizeType32 const* generationLengths, float const* posteriorThresholds, float const* posteriorAlphas,
    float const* temperatures, SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 maxDecodingTokens,
    SizeType32 vocabSize, hipStream_t stream);
template void maskLogitsBasedOnEntropy(half** logitsPtrs, TokenIdType** outputIdsPtrs, bool* skipDecode,
    TokenIdType* outputIds, float* runtimeMultinomialTopP, half const* probsPtr, float const* entropies,
    SizeType32 const* generationLengths, float const* posteriorThresholds, float const* posteriorAlphas,
    float const* temperatures, SizeType32 const* batchSlots, SizeType32 batchSize, SizeType32 maxDecodingTokens,
    SizeType32 vocabSize, hipStream_t stream);

template <typename T>
void typicalAcceptanceSampling(TypicalAcceptanceSampling<T> const& params, hipStream_t stream)
{
    int8_t* workspaceBytePtr = reinterpret_cast<int8_t*>(params.workspace);
    size_t offset{0};

    int8_t* samplingWorkspace
        = reinterpret_cast<int8_t*>(tensorrt_llm::common::nextWorkspacePtr(workspaceBytePtr, offset,
            tensorrt_llm::kernels::getAirTopPWorkspaceSize<T>(
                params.batchSize * params.maxDecodingTokens, params.vocabSize, /* isDeterministic */ true)));

    float* entropy = reinterpret_cast<float*>(tensorrt_llm::common::nextWorkspacePtr(
        workspaceBytePtr, offset, params.batchSize * params.maxDecodingTokens * sizeof(float)));

    float* runtimeMultinomialTopP = reinterpret_cast<float*>(tensorrt_llm::common::nextWorkspacePtr(
        workspaceBytePtr, offset, params.batchSize * params.maxDecodingTokens * sizeof(float)));

    T* probs = reinterpret_cast<T*>(tensorrt_llm::common::nextWorkspacePtr(
        workspaceBytePtr, offset, params.batchSize * params.maxDecodingTokens * params.vocabSize * sizeof(T)));

    TokenIdType** outputIdsPtrs = reinterpret_cast<TokenIdType**>(tensorrt_llm::common::nextWorkspacePtr(
        workspaceBytePtr, offset, params.batchSize * params.maxDecodingTokens * sizeof(TokenIdType*)));

    bool* skipDecodePtr = reinterpret_cast<bool*>(tensorrt_llm::common::nextWorkspacePtr(
        workspaceBytePtr, offset, params.batchSize * params.maxDecodingTokens * sizeof(bool)));

    // compute probs and entropy
    {
        BiasSoftmaxParams<T> biasSoftmaxParams;
        biasSoftmaxParams.logitsPtrs = params.logitsPtrs;
        biasSoftmaxParams.probs = probs;
        biasSoftmaxParams.outputEntropy = entropy;
        biasSoftmaxParams.temperatures = params.temperatures;
        biasSoftmaxParams.beamWidths = params.generationLengths;
        biasSoftmaxParams.batchSlots = params.batchSlots;
        biasSoftmaxParams.batchSize = params.batchSize;
        biasSoftmaxParams.maxBatchSize = params.batchSize;
        biasSoftmaxParams.maxBeamWidth = params.maxDecodingTokens;
        biasSoftmaxParams.vocabSize = params.vocabSize;
        biasSoftmaxParams.vocabSizePadded = params.vocabSize;
        biasSoftmaxParams.skipSoftMax = false;
        biasSoftmaxParams.batchSlotsLogits = false;
        biasSoftmaxParams.ptrsForBeams = true;

        biasSoftmaxParams.checkParams();

        invokeAddBiasSoftMax(biasSoftmaxParams, stream);

        sync_check_cuda_error(stream);
    }

    // correct logits based on the probs and entropy
    {
        maskLogitsBasedOnEntropy(params.logitsPtrs, outputIdsPtrs, skipDecodePtr, params.outputIds,
            runtimeMultinomialTopP, probs, entropy, params.generationLengths, params.posteriorThresholds,
            params.posteriorAlphas, params.temperatures, params.batchSlots, params.batchSize, params.maxDecodingTokens,
            params.vocabSize, stream);

        sync_check_cuda_error(stream);
    }

    // compute probs of the corrected logits
    {
        BiasSoftmaxParams<T> biasSoftmaxParams;
        biasSoftmaxParams.logitsPtrs = params.logitsPtrs;
        biasSoftmaxParams.probs = probs;
        biasSoftmaxParams.beamWidths = params.generationLengths;
        biasSoftmaxParams.batchSlots = params.batchSlots;
        biasSoftmaxParams.batchSize = params.batchSize;
        biasSoftmaxParams.maxBatchSize = params.batchSize;
        biasSoftmaxParams.maxBeamWidth = params.maxDecodingTokens;
        biasSoftmaxParams.vocabSize = params.vocabSize;
        biasSoftmaxParams.vocabSizePadded = params.vocabSize;
        biasSoftmaxParams.skipSoftMax = false;
        biasSoftmaxParams.batchSlotsLogits = false;
        biasSoftmaxParams.ptrsForBeams = true;

        biasSoftmaxParams.checkParams();

        invokeAddBiasSoftMax(biasSoftmaxParams, stream);

        sync_check_cuda_error(stream);
    }

    // do multinomial sampling
    {
        TopPSamplingKernelParams<T> samplingParams{};
        samplingParams.probs = probs;
        samplingParams.outputIdsPtrs = outputIdsPtrs;
        samplingParams.workspace = samplingWorkspace;
        samplingParams.topPs = runtimeMultinomialTopP;
        samplingParams.batchSlots = params.batchSlots;
        samplingParams.hiprandState = params.curandStats;
        samplingParams.randomVals = params.randomVals;
        samplingParams.skipDecode = skipDecodePtr;

        samplingParams.batchSize = params.batchSize * params.maxDecodingTokens;
        samplingParams.maxBatchSize = params.batchSize * params.maxDecodingTokens;
        samplingParams.vocabSizePadded = params.vocabSize;
        samplingParams.maxSeqLen = 1;

        samplingParams.blockNum
            = calcAirTopPBlockNum<T>(samplingParams.batchSize, samplingParams.vocabSizePadded, params.smCnt, true);
        samplingParams.isDeterministic = true;

        samplingParams.checkParams();

        tensorrt_llm::kernels::invokeBatchAirTopPSampling<T>(samplingParams, stream);

        sync_check_cuda_error(stream);
    }
}

template void typicalAcceptanceSampling(TypicalAcceptanceSampling<float> const& params, hipStream_t stream);
template void typicalAcceptanceSampling(TypicalAcceptanceSampling<half> const& params, hipStream_t stream);

template <typename T>
size_t getTypicalAcceptanceWorkspaceSize(SizeType32 batchSize, SizeType32 maxDecodingTokens, SizeType32 vocabSizePadded)
{
    SizeType32 constexpr NUM_BUFFERS{6};
    size_t workspaces[NUM_BUFFERS];
    workspaces[0] = tensorrt_llm::kernels::getAirTopPWorkspaceSize<T>(
        batchSize * maxDecodingTokens, vocabSizePadded, /* isDeterministic */ true);
    // entropy
    workspaces[1] = batchSize * maxDecodingTokens * sizeof(float);
    // runtimeMultinomialTopP
    workspaces[2] = batchSize * maxDecodingTokens * sizeof(float);
    // probs
    workspaces[3] = batchSize * maxDecodingTokens * vocabSizePadded * sizeof(T);
    // outputIdsPtrs
    workspaces[4] = batchSize * maxDecodingTokens * sizeof(TokenIdType*);
    // skipDecode
    workspaces[5] = batchSize * maxDecodingTokens * sizeof(bool);
    auto const workspaceSize = tensorrt_llm::common::calculateTotalWorkspaceSize(workspaces, NUM_BUFFERS);
    return workspaceSize;
}

template size_t getTypicalAcceptanceWorkspaceSize<float>(
    SizeType32 batchSize, SizeType32 maxDecodingTokens, SizeType32 vocabSizePadded);
template size_t getTypicalAcceptanceWorkspaceSize<half>(
    SizeType32 batchSize, SizeType32 maxDecodingTokens, SizeType32 vocabSizePadded);

} // namespace tensorrt_llm::kernels::speculative_decoding
