#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "moeTopKFuncs.cuh"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/kernels/archCondition.h"
#include "tensorrt_llm/kernels/customMoeRoutingKernels.h"
#include <climits> // For INT_MAX
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <hipcub/hipcub.hpp>
#include <cuda/std/limits> // For numeric_limits
#include <math.h>

namespace cg = cooperative_groups;
using namespace tensorrt_llm::common;

namespace tensorrt_llm::kernels
{

static constexpr int BLOCK_SIZE = 1024;
static constexpr int WARP_SIZE = 32;
static constexpr int WARPS_PER_BLOCK = BLOCK_SIZE / WARP_SIZE;

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename T>
__device__ T calcSoftmax(cg::thread_block_tile<WARP_SIZE> const& warp, T score, int32_t laneIdx, int32_t NumTopExperts)
{
    T maxScore = T{-INFINITY};
    if (laneIdx < NumTopExperts)
    {
        maxScore = score >= maxScore ? score : maxScore;
    }
    maxScore = cg::reduce(warp, maxScore, cg::greater<T>());

    float sumScore{0.f};
    float newScore;
    // Get the summation of scores for each token
    if (laneIdx < NumTopExperts)
    {
        newScore = static_cast<float>(score) - static_cast<float>(maxScore);
        newScore = static_cast<float>(exp(newScore));
        sumScore += newScore;
    }
    sumScore = cg::reduce(warp, sumScore, cg::plus<float>());

    if (laneIdx < NumTopExperts)
    {
        score = static_cast<T>(newScore / sumScore);
    }

    return score;
}

template <typename DataType, int VecSize>
__device__ void calcSoftmax(cg::thread_block_tile<WARP_SIZE> const& warp, DataType (&scores)[VecSize])
{
    DataType maxScore = DataType{-INFINITY};
    DataType sumScore = DataType{0.f};

    // Get the max score for each token
#pragma unroll
    for (int i = 0; i < VecSize; ++i)
    {
        maxScore = scores[i] >= maxScore ? scores[i] : maxScore;
    }
    maxScore = cg::reduce(warp, maxScore, cg::greater<DataType>());

    // Get the summation of scores for each token
#pragma unroll
    for (int i = 0; i < VecSize; ++i)
    {
        scores[i] = static_cast<DataType>(exp(scores[i] - maxScore));
        sumScore += scores[i];
    }
    sumScore = cg::reduce(warp, sumScore, cg::plus<DataType>());

    // Normalize the scores
#pragma unroll
    for (int i = 0; i < VecSize; ++i)
    {
        scores[i] = static_cast<DataType>(scores[i] / sumScore);
    }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename InputT, typename OutputT, typename IdxT, int MaxNumExperts, int MaxNumTopExperts,
    bool DoSoftmaxBeforeTopK>
__global__ void customMoeRoutingKernel(InputT* routerLogits, OutputT* topkValues, IdxT* topkIndices,
    int32_t const numTokens, int32_t const numExperts, int32_t const topK)
{
    using BaseType = std::conditional_t<DoSoftmaxBeforeTopK, float, InputT>;
    uint32_t const blockRank = blockIdx.x;
    uint32_t const tIdx = BLOCK_SIZE * blockRank + threadIdx.x;
    uint32_t const warpIdx = tIdx / WARP_SIZE;
    uint32_t const laneIdx = tIdx % WARP_SIZE;
    uint32_t const warpNum = gridDim.x * WARPS_PER_BLOCK;
    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<WARP_SIZE>(block);

    BaseType minScore = BaseType{-INFINITY};
    for (uint32_t tokenId = warpIdx; tokenId < numTokens; tokenId += warpNum)
    {
        auto scoreOffset = tokenId * numExperts;
        auto outputOffset = tokenId * topK;

        BaseType inputScore[MaxNumExperts / WARP_SIZE];
        IdxT inputIndex[MaxNumExperts / WARP_SIZE];

        BaseType warpTopKScore[MaxNumTopExperts];
        IdxT warpTopKExpertIdx[MaxNumTopExperts];

        // Load scores and indices for this warp
        for (uint32_t i = 0; i < MaxNumExperts / WARP_SIZE; ++i)
        {
            auto expertIdx = i * WARP_SIZE + laneIdx;
            inputScore[i]
                = expertIdx < numExperts ? static_cast<BaseType>(routerLogits[scoreOffset + expertIdx]) : minScore;
            inputIndex[i] = expertIdx;
        }

        if constexpr (DoSoftmaxBeforeTopK)
        {
            calcSoftmax(warp, inputScore);
        }
        // Reduce topK scores and indices for this warp
        reduce_topk::reduceTopK(warp, warpTopKScore, warpTopKExpertIdx, inputScore, inputIndex, minScore);

        // Normalize the scores
        if constexpr (DoSoftmaxBeforeTopK)
        {
            if (laneIdx < topK)
            {
                topkValues[outputOffset + laneIdx] = static_cast<OutputT>(warpTopKScore[laneIdx]);
                topkIndices[outputOffset + laneIdx] = warpTopKExpertIdx[laneIdx];
            }
        }
        else
        {
            auto softmaxScore = calcSoftmax(warp,
                laneIdx < topK ? static_cast<float>(warpTopKScore[laneIdx]) : static_cast<float>(minScore), laneIdx,
                topK);
            if (laneIdx < topK)
            {
                topkValues[outputOffset + laneIdx] = static_cast<OutputT>(softmaxScore);
                topkIndices[outputOffset + laneIdx] = warpTopKExpertIdx[laneIdx];
            }
        }
    } // end for tokenId
}

int nextPowerOfTwo(int num)
{
    if (num <= 0)
    {
        return 1; // Handle invalid input
    }
    int power = 1;
    while (power < num)
    {
        // Check for overflow before shifting
        if (power > INT_MAX / 2)
        {
            return power;
        }
        power <<= 1;
    }
    return power;
}

#define CASE(MAX_NUM_EXPERTS)                                                                                          \
    case MAX_NUM_EXPERTS:                                                                                              \
        switch (maxNumTopExperts)                                                                                      \
        {                                                                                                              \
        case 1:                                                                                                        \
            kernelInstance = &customMoeRoutingKernel<InputT, OutputT, IdxT, MAX_NUM_EXPERTS, 1, DoSoftmaxBeforeTopK>;  \
            break;                                                                                                     \
        case 2:                                                                                                        \
            kernelInstance = &customMoeRoutingKernel<InputT, OutputT, IdxT, MAX_NUM_EXPERTS, 2, DoSoftmaxBeforeTopK>;  \
            break;                                                                                                     \
        case 4:                                                                                                        \
            kernelInstance = &customMoeRoutingKernel<InputT, OutputT, IdxT, MAX_NUM_EXPERTS, 4, DoSoftmaxBeforeTopK>;  \
            break;                                                                                                     \
        case 8:                                                                                                        \
            kernelInstance = &customMoeRoutingKernel<InputT, OutputT, IdxT, MAX_NUM_EXPERTS, 8, DoSoftmaxBeforeTopK>;  \
            break;                                                                                                     \
        default: kernelInstance = nullptr; break;                                                                      \
        }                                                                                                              \
        break;

template <typename InputT, typename OutputT, typename IdxT, bool DoSoftmaxBeforeTopK>
void invokeRenormMoeRouting(InputT* routerLogits, OutputT* topkValues, IdxT* topkIndices, int64_t const numTokens,
    int64_t const numExperts, int64_t const topK, hipStream_t const stream)
{

    const uint32_t maxNumBlocks = 1024;
    const uint32_t numBlocks = std::min(static_cast<uint32_t>((numTokens - 1) / WARPS_PER_BLOCK + 1), maxNumBlocks);

    uint32_t maxNumExperts = nextPowerOfTwo(numExperts) < 32 ? 32 : nextPowerOfTwo(numExperts);
    uint32_t maxNumTopExperts = nextPowerOfTwo(topK);

    auto* kernelInstance = &customMoeRoutingKernel<InputT, OutputT, IdxT, 128, 8, DoSoftmaxBeforeTopK>;

    switch (maxNumExperts)
    {
        CASE(32)
        CASE(64)
        CASE(96)
        CASE(128)
    default: kernelInstance = nullptr; break;
    }

    if (kernelInstance == nullptr)
    {
        TLLM_CHECK_WITH_INFO(kernelInstance != nullptr, "Can not find corresponding kernel instance.");
    }

    dim3 renormMoeRoutingGridDim(numBlocks);
    dim3 renormMoeRoutingBlockDim(BLOCK_SIZE);
    hipLaunchConfig_t config;
    config.gridDim = renormMoeRoutingGridDim;
    config.blockDim = renormMoeRoutingBlockDim;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    cudaLaunchKernelEx(&config, kernelInstance, routerLogits, topkValues, topkIndices, static_cast<int32_t>(numTokens),
        static_cast<int32_t>(numExperts), static_cast<int32_t>(topK));
    sync_check_cuda_error(stream);
}

#define INSTANTIATE_RENORM_MOE_ROUTING(InputT, OutputT, IdxT, DoSoftmaxBeforeTopK)                                     \
    template void invokeRenormMoeRouting<InputT, OutputT, IdxT, DoSoftmaxBeforeTopK>(InputT * routerLogits,            \
        OutputT * topkValues, IdxT * topkIndices, int64_t const numTokens, int64_t const numExperts,                   \
        int64_t const topK, hipStream_t const stream);

INSTANTIATE_RENORM_MOE_ROUTING(float, float, int32_t, false);
INSTANTIATE_RENORM_MOE_ROUTING(half, float, int32_t, false);
#ifdef ENABLE_BF16
INSTANTIATE_RENORM_MOE_ROUTING(__hip_bfloat16, float, int32_t, false);
#endif

INSTANTIATE_RENORM_MOE_ROUTING(float, float, int32_t, true);
INSTANTIATE_RENORM_MOE_ROUTING(half, float, int32_t, true);
#ifdef ENABLE_BF16
INSTANTIATE_RENORM_MOE_ROUTING(__hip_bfloat16, float, int32_t, true);
#endif

} // namespace tensorrt_llm::kernels
