#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "fp8_blockscale_gemm.h"
#include "fp8_blockscale_gemm_kernel.cuh"
#include "tensorrt_llm/common/logger.h"

namespace tensorrt_llm::kernels::fp8_blockscale_gemm
{

template <typename ElementA, typename ElementB, typename ElementD>
CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::CutlassFp8BlockScaleGemmRunner()
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
}

template <typename ElementA, typename ElementB, typename ElementD>
CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::~CutlassFp8BlockScaleGemmRunner()
{
    TLLM_LOG_DEBUG(__PRETTY_FUNCTION__);
}

template <typename ElementA, typename ElementB, typename ElementD>
void CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::gemm(void* mat_d, void const* mat_a,
    void const* mat_b, int shape_m, int shape_n, int shape_k, hipStream_t stream, float const* scales_a,
    float const* scales_b)
{
    constexpr bool internal_quantize_a = !std::is_same_v<ElementA, __hip_fp8_e4m3_fnuz>;
    constexpr bool internal_quantize_b = !std::is_same_v<ElementB, __hip_fp8_e4m3_fnuz>;
    __hip_fp8_e4m3_fnuz* fp8_mat_a;
    __hip_fp8_e4m3_fnuz* fp8_mat_b;
    float* per_token_per_128c_scales;
    float* per_block_scales;

    auto* ws_ptr = workspace_;
    if constexpr (internal_quantize_a || internal_quantize_b)
    {
        TLLM_CHECK(ws_ptr != nullptr);
    }

    if constexpr (internal_quantize_a)
    {
        fp8_mat_a = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(ws_ptr);
        ws_ptr += max_shape_m_4_align_ * shape_k * sizeof(__hip_fp8_e4m3_fnuz);
        per_token_per_128c_scales = reinterpret_cast<float*>(ws_ptr);
        ws_ptr += max_shape_m_4_align_ * div_up(shape_k, 128) * sizeof(float);
    }

    if constexpr (internal_quantize_b)
    {
        fp8_mat_b = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(ws_ptr);
        ws_ptr += shape_n * shape_k * sizeof(__hip_fp8_e4m3_fnuz);
        per_block_scales = reinterpret_cast<float*>(ws_ptr);
        ws_ptr += div_up(shape_n, 128) * div_up(shape_k, 128) * sizeof(float);
    }

#ifdef COMPILE_HOPPER_TMA_GEMMS
    if constexpr (internal_quantize_a && internal_quantize_b)
    {
        fp8_gemm_run(reinterpret_cast<__hip_bfloat16 const*>(mat_a), fp8_mat_a, shape_k, per_token_per_128c_scales,
            reinterpret_cast<__hip_bfloat16 const*>(mat_b), fp8_mat_b, shape_k, per_block_scales,
            reinterpret_cast<__hip_bfloat16*>(mat_d), shape_n, shape_m, shape_n, shape_k, stream, internal_quantize_a,
            internal_quantize_b);
    }

    if constexpr (internal_quantize_a && !internal_quantize_b)
    {
        fp8_gemm_run(reinterpret_cast<__hip_bfloat16 const*>(mat_a), fp8_mat_a, shape_k, per_token_per_128c_scales,
            nullptr, reinterpret_cast<__hip_fp8_e4m3_fnuz*>(const_cast<void*>(mat_b)), shape_k, const_cast<float*>(scales_b),
            reinterpret_cast<__hip_bfloat16*>(mat_d), shape_n, shape_m, shape_n, shape_k, stream, internal_quantize_a,
            internal_quantize_b);
    }
#else  // COMPILE_HOPPER_TMA_GEMMS
    TLLM_THROW("fp8 blockscale gemm only support Hopper.");
#endif // COMPILE_HOPPER_TMA_GEMMS
}

template <typename ElementA, typename ElementB, typename ElementD>
void CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::gemm(__hip_fp8_e4m3_fnuz const* mat_a, int ld_a,
    __hip_fp8_e4m3_fnuz const* mat_b, int ld_b, __hip_bfloat16* mat_d, int ld_d, int shape_m, int shape_n, int shape_k,
    float const* scales_a, float const* scales_b, hipStream_t stream)
{

    fp8_gemm_run(const_cast<__hip_fp8_e4m3_fnuz*>(mat_a), ld_a, const_cast<__hip_fp8_e4m3_fnuz*>(mat_b), ld_b, mat_d, ld_d, shape_m,
        shape_n, shape_k, const_cast<float*>(scales_a), const_cast<float*>(scales_b), stream);
}

template <typename ElementA, typename ElementB, typename ElementD>
void CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::moeGemm(void* mat_d, void const* mat_a,
    void const* mat_b, int64_t const* problem_m_offsets, size_t num_problems, size_t shape_n, size_t shape_k,
    hipStream_t stream, float const* scales_a, float const* scales_b)
{
    constexpr bool internal_quantize_a = !std::is_same_v<ElementA, __hip_fp8_e4m3_fnuz>;
    constexpr bool internal_quantize_b = !std::is_same_v<ElementB, __hip_fp8_e4m3_fnuz>;

    __hip_fp8_e4m3_fnuz* fp8_mat_a;
    float* per_token_per_128c_scales;
    __hip_fp8_e4m3_fnuz* fp8_mat_b;
    float* per_block_scales;
    int64_t* problem_m_padded_offsets;

    auto* ws_ptr = workspace_;
    if constexpr (internal_quantize_a || internal_quantize_b)
    {
        TLLM_CHECK(ws_ptr != nullptr);
    }

    if constexpr (internal_quantize_a)
    {
        fp8_mat_a = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(ws_ptr);
        ws_ptr += max_shape_m_4_align_ * shape_k * sizeof(__hip_fp8_e4m3_fnuz);
        per_token_per_128c_scales = reinterpret_cast<float*>(ws_ptr);
        ws_ptr += max_shape_m_32_align_padded_ * div_up(shape_k, 128) * sizeof(float);
    }
    else
    {
        fp8_mat_a = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(const_cast<void*>(mat_a));
        per_token_per_128c_scales = const_cast<float*>(scales_a);
    }

    if constexpr (internal_quantize_b)
    {
        fp8_mat_b = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(ws_ptr);
        ws_ptr += num_problems * shape_n * shape_k * sizeof(__hip_fp8_e4m3_fnuz);
        per_block_scales = reinterpret_cast<float*>(ws_ptr);
    }
    else
    {
        for (int i = 0; i < num_problems; i++)
        {
            fp8_mat_b = reinterpret_cast<__hip_fp8_e4m3_fnuz*>(const_cast<void*>(mat_b));
            per_block_scales = const_cast<float*>(scales_b);
        }
    }

    problem_m_padded_offsets = reinterpret_cast<int64_t*>(ws_ptr);
    ws_ptr += (num_problems + 1) * sizeof(int64_t);

#ifdef COMPILE_HOPPER_TMA_GEMMS
    if constexpr (std::is_same_v<ElementA, __hip_bfloat16> && std::is_same_v<ElementB, __hip_bfloat16>)
    {
        fp8_grouped_gemm_run(reinterpret_cast<__hip_bfloat16 const*>(mat_a), fp8_mat_a, per_token_per_128c_scales,
            reinterpret_cast<__hip_bfloat16 const*>(mat_b), fp8_mat_b, per_block_scales,
            reinterpret_cast<__hip_bfloat16*>(mat_d), problem_m_offsets, problem_m_padded_offsets, num_problems,
            expected_m_, max_shape_m_4_align_, max_shape_m_32_align_padded_, shape_n, shape_k, stream,
            internal_quantize_a, internal_quantize_b);
    }
    else if constexpr (std::is_same_v<ElementA, __hip_bfloat16> && std::is_same_v<ElementB, __hip_fp8_e4m3_fnuz>)
    {
        fp8_grouped_gemm_run(reinterpret_cast<__hip_bfloat16 const*>(mat_a), fp8_mat_a, per_token_per_128c_scales,
            nullptr, fp8_mat_b, per_block_scales, reinterpret_cast<__hip_bfloat16*>(mat_d), problem_m_offsets,
            problem_m_padded_offsets, num_problems, expected_m_, max_shape_m_4_align_, max_shape_m_32_align_padded_,
            shape_n, shape_k, stream, internal_quantize_a, internal_quantize_b);
    }
    else if constexpr (std::is_same_v<ElementA, __hip_fp8_e4m3_fnuz> && std::is_same_v<ElementB, __hip_fp8_e4m3_fnuz>)
    {
        fp8_grouped_gemm_run(nullptr, fp8_mat_a, per_token_per_128c_scales,
            reinterpret_cast<__hip_bfloat16 const*>(mat_b), fp8_mat_b, per_block_scales,
            reinterpret_cast<__hip_bfloat16*>(mat_d), problem_m_offsets, problem_m_padded_offsets, num_problems,
            expected_m_, max_shape_m_4_align_, max_shape_m_32_align_padded_, shape_n, shape_k, stream,
            internal_quantize_a, internal_quantize_b);
    }
    else
    {
        TLLM_THROW("fp8 blockscale gemm only support __hip_fp8_e4m3_fnuz or bfloat16 as dataType.");
    }
#else
    TLLM_THROW("fp8 blockscale gemm only support Hopper.");
#endif
}

template <typename ElementA, typename ElementB, typename ElementD>
void CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::strideBatchGemm(__hip_bfloat16* mat_d, int ld_d,
    int stride_d, __hip_fp8_e4m3_fnuz* mat_a, int ld_a, int stride_a, __hip_fp8_e4m3_fnuz* mat_b, int ld_b, int stride_b,
    int num_problems, int shape_m, int shape_n, int shape_k, hipStream_t stream, float* scales_a, int stride_scales_a,
    float* scales_b)
{

    fp8_stride_batch_gemm_run(nullptr, mat_a, scales_a, ld_a, stride_a, stride_scales_a, nullptr, mat_b, scales_b, ld_b,
        stride_b, mat_d, ld_d, stride_d, num_problems, shape_m, shape_n, shape_k, stream, false, false);
}

template <typename ElementA, typename ElementB, typename ElementD>
void CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::fp8CS1x128(
    __hip_fp8_e4m3_fnuz* mat_quant, float* scales, __hip_bfloat16 const* mat, int shape_x, int shape_y, hipStream_t stream)
{
    fp8_1x128_cs(mat_quant, scales, mat, shape_x, shape_y, stream);
}

template <typename ElementA, typename ElementB, typename ElementD>
void CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::fp8CS1x128Reshape(__hip_fp8_e4m3_fnuz* mat_quant,
    float* scales, __hip_bfloat16 const* mat, int shape_x, int shape_h, int shape_y, int stride_x, hipStream_t stream)
{
    fp8_1x128_cs_reshape(mat_quant, scales, mat, shape_x, shape_h, shape_y, stride_x, stream);
}

template <typename ElementA, typename ElementB, typename ElementD>
void CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::fp8CS128x128(
    __hip_fp8_e4m3_fnuz* mat_quant, float* scales, __hip_bfloat16 const* mat, int shape_x, int shape_y, hipStream_t stream)
{
    fp8_128x128_cs(mat_quant, scales, mat, shape_x, shape_y, stream);
}

template <typename ElementA, typename ElementB, typename ElementD>
size_t CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::getWorkspaceSizeBase(
    size_t max_shape_m, size_t shape_n, size_t shape_k, size_t num_problems)
{
    max_shape_m_4_align_ = std::max(max_shape_m_4_align_, int64_t(div_up(max_shape_m, 4) * 4));
    if (expected_m_ == 0)
    {
        expected_m_ = div_up(max_shape_m_4_align_, num_problems);
    }
    max_shape_m_32_align_padded_ = int64_t(div_up(max_shape_m + num_problems * 31, 32) * 32);

    constexpr bool internal_quantize_a = !std::is_same_v<ElementA, __hip_fp8_e4m3_fnuz>;
    constexpr bool internal_quantize_b = !std::is_same_v<ElementB, __hip_fp8_e4m3_fnuz>;
    size_t total_workspace_size = 0;
    if constexpr (internal_quantize_a)
    {
        // fp8_mat_a
        total_workspace_size += max_shape_m_4_align_ * shape_k * sizeof(__hip_fp8_e4m3_fnuz);
        // scales_a
        total_workspace_size += max_shape_m_32_align_padded_ * div_up(shape_k, 128) * sizeof(float);
    }

    if constexpr (internal_quantize_b)
    {
        // fp8_mat_b
        total_workspace_size += num_problems * shape_n * shape_k * sizeof(__hip_fp8_e4m3_fnuz);
        // scales_b
        total_workspace_size += num_problems * div_up(shape_k, 128) * div_up(shape_n, 128) * sizeof(float);
    }

    total_workspace_size += (num_problems + 1) * sizeof(int64_t);

    return total_workspace_size;
}

template <typename ElementA, typename ElementB, typename ElementD>
size_t CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::getWorkspaceSize(
    size_t shape_m, size_t shape_n, size_t shape_k, size_t top_k, size_t num_problems)
{
    expected_m_ = shape_m;
    return getWorkspaceSizeBase(shape_m * top_k, shape_n, shape_k, num_problems);
}

template <typename ElementA, typename ElementB, typename ElementD>
size_t CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::getFP8DataSize(
    int shape_m, int shape_n, bool is_act)
{
    int shape_m_4_align = div_up(shape_m, 4) * 4;
    constexpr bool internal_quantize_a = !std::is_same_v<ElementA, __hip_fp8_e4m3_fnuz>;
    constexpr bool internal_quantize_b = !std::is_same_v<ElementB, __hip_fp8_e4m3_fnuz>;
    if (is_act && internal_quantize_a)
    {
        return div_up(shape_m_4_align * shape_n * sizeof(__hip_fp8_e4m3_fnuz), 128) * 128;
    }

    if ((!is_act) && internal_quantize_b)
    {
        return div_up(shape_m * shape_n * sizeof(__hip_fp8_e4m3_fnuz), 128) * 128;
    }
    return 0;
}

template <typename ElementA, typename ElementB, typename ElementD>
size_t CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::getActScaleSize(int shape_m, int shape_k)
{
    int shape_m_4_align = div_up(shape_m, 4) * 4;
    constexpr bool internal_quantize_a = !std::is_same_v<ElementA, __hip_fp8_e4m3_fnuz>;
    size_t total_workspace_size = 0;
    if constexpr (internal_quantize_a)
    {
        // scales_a
        total_workspace_size += div_up(shape_m_4_align * div_up(shape_k, 128) * sizeof(float), 128) * 128;
    }
    return total_workspace_size;
}

template <typename ElementA, typename ElementB, typename ElementD>
size_t CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::getWeightScaleSize(int shape_n, int shape_k)
{
    constexpr bool internal_quantize_b = !std::is_same_v<ElementB, __hip_fp8_e4m3_fnuz>;
    size_t total_workspace_size = 0;
    if constexpr (internal_quantize_b)
    {
        // scales_b
        total_workspace_size += div_up(div_up(shape_k, 128) * div_up(shape_n, 128) * sizeof(float), 128) * 128;
    }

    return total_workspace_size;
}

template <typename ElementA, typename ElementB, typename ElementD>
size_t CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::getActWorkspaceSize(int shape_m, int shape_k)
{
    return getFP8DataSize(shape_m, shape_k, true) + getActScaleSize(shape_m, shape_k);
}

template <typename ElementA, typename ElementB, typename ElementD>
size_t CutlassFp8BlockScaleGemmRunner<ElementA, ElementB, ElementD>::getWeightWorkspaceSize(int shape_n, int shape_k)
{
    return getFP8DataSize(shape_n, shape_k, false) + getWeightScaleSize(shape_n, shape_k);
}

template class CutlassFp8BlockScaleGemmRunner<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16>;
template class CutlassFp8BlockScaleGemmRunner<__hip_bfloat16, __hip_fp8_e4m3_fnuz, __hip_bfloat16>;
template class CutlassFp8BlockScaleGemmRunner<__hip_fp8_e4m3_fnuz, __hip_bfloat16, __hip_bfloat16>;
template class CutlassFp8BlockScaleGemmRunner<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16>;

} // namespace tensorrt_llm::kernels::fp8_blockscale_gemm
