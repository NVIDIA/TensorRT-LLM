#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: NVIDIA TensorRT Source Code License Agreement
 *
 * NVIDIA CORPORATION, its affiliates and licensors retain all intellectual
 * property and proprietary rights in and to this material, related
 * documentation and any modifications thereto. Any use, reproduction,
 * disclosure or distribution of this material and related documentation
 * without an express license agreement from NVIDIA CORPORATION or
 * its affiliates is strictly prohibited.
 */

#include "moe_gemm_template_dispatch.h"

namespace tensorrt_llm::kernels::cutlass_kernels
{
#ifdef ENABLE_BF16
template class MoeGemmRunner<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16>;
#endif
} // namespace tensorrt_llm::kernels::cutlass_kernels
