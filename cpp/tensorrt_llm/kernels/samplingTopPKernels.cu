#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/samplingTopKKernels.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"

using namespace tensorrt_llm::common;
using namespace tensorrt_llm::runtime;

namespace tensorrt_llm::kernels
{
__global__ void topPInitialize(TokenIdType* topPIdValBuf, SizeType32* topPOffsetBuf, SizeType32* beginTopPOffsetBuf,
    SizeType32 batchSize, SizeType32 vocabSize)
{
    auto const tid = static_cast<SizeType32>(threadIdx.x);
    auto const bid = static_cast<SizeType32>(blockIdx.x);

    if (bid == 0)
    {
        for (auto i = tid; i < batchSize + 1; i += static_cast<SizeType32>(blockDim.x))
        {
            // Inclusive sum of offsets to vocab rows
            topPOffsetBuf[i] = i * vocabSize;
            beginTopPOffsetBuf[i] = topPOffsetBuf[i];
        }
    }

    auto index = tid + bid * static_cast<SizeType32>(blockDim.x);

    while (index < batchSize * vocabSize)
    {
        // Set value at {bi, vi} position to vi
        topPIdValBuf[index] = index % vocabSize;
        index += static_cast<SizeType32>(blockDim.x * gridDim.x);
    }
}

void invokeTopPInitialize(TokenIdType* topPIdValBuf, SizeType32* topPOffsetBuf, SizeType32* beginTopPOffsetBuf,
    SizeType32 batchSize, SizeType32 vocabSize, hipStream_t stream)
{
    // vocabSize: the column number of logits_buffer for top_p sampling
    // TODO(nkorobov): launch based on available resources
    topPInitialize<<<32, 512, 0, stream>>>(topPIdValBuf, topPOffsetBuf, beginTopPOffsetBuf, batchSize, vocabSize);
}

template <typename T, int THREADBLOCK_SIZE>
__launch_bounds__(THREADBLOCK_SIZE) __global__ void topPBeamTopKKernel(T const* probs, // prob.
    TokenIdType* topKTmpIdBuf, T* topKTmpValBuf, FinishedState const* finishedInput, SizeType32 vocabSize,
    SizeType32* offsetBuf, SizeType32* beginOffsetBuf, float const* topPs, bool const* skipDecode,
    SizeType32 const* batchSlots)
{
    /**
     * Kernel performs top 1 search and saves the token with largest probability if it exceeds probability threshold
     */
    SizeType32 constexpr MAX_K = 1;
    auto const threadId = static_cast<SizeType32>(threadIdx.x);
    auto const batchId = static_cast<SizeType32>(blockIdx.x);
    auto const batchSlot = batchSlots[batchId];

    // Skip decoding kernel if configured
    if ((skipDecode != nullptr && skipDecode[batchSlot])
        || (finishedInput != nullptr && finishedInput[batchSlot].isSkipDecoding()))
    {
        // Required to skip radix sort
        beginOffsetBuf[batchId] += vocabSize;
        return;
    }

    float pThreshold = topPs[batchSlot];

    typedef hipcub::BlockReduce<TopK<T, MAX_K>, THREADBLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    TopK<T, MAX_K> partial;

    bool const IS_FP16 = std::is_same<T, half>::value;
    T const MAX_T_VAL = (IS_FP16) ? HALF_FLT_MAX : FLT_MAX;

#pragma unroll
    for (SizeType32 i = 0; i < MAX_K; ++i)
    {
        partial.p[i] = -1;
        partial.u[i] = -MAX_T_VAL;
    }

#pragma unroll
    for (SizeType32 elemId = static_cast<SizeType32>(threadId); elemId < vocabSize; elemId += THREADBLOCK_SIZE)
    {
        auto index = elemId + batchId * vocabSize;
        partial.insert(probs[index], elemId);
    }

    TopK<T, MAX_K> total = BlockReduce(temp_storage).Reduce(partial, reduce_topk_op<T, MAX_K>);

    if (threadId == 0)
    {
        beginOffsetBuf[batchId] = offsetBuf[batchId];
        T sumProb = (T) (0.0f);

#pragma unroll
        for (SizeType32 i = 0; i < MAX_K; i++)
        {
            sumProb += total.u[i];
        }

        if ((float) sumProb >= pThreshold)
        {
            beginOffsetBuf[batchId] += vocabSize;
            auto index = batchId * vocabSize;

#pragma unroll
            for (SizeType32 i = 0; i < MAX_K; ++i)
            {
                topKTmpIdBuf[index + i] = total.p[i];
                topKTmpValBuf[index + i] = total.u[i];
            }
        }
    }
}

struct BlockPrefixCallbackOp
{
    // Running prefix
    float running_total;

    // Constructor
    __device__ BlockPrefixCallbackOp(float running_total)
        : running_total(running_total)
    {
    }

    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide
    // scan.
    __device__ float operator()(float block_aggregate)
    {
        float old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

template <typename T>
__device__ void epilogue(SizeType32 batchId, SizeType32 currentStep, SizeType32 offset, TokenIdType** ids,
    TokenIdType const* sortedIdVals, T const* sortedProbs, float* cumLogProbs, float* outputLogProbs,
    TokenIdType const* endIds, SizeType32* sequenceLengths, FinishedState* finishedOutput, SizeType32 maxBatchSize)
{
    ids[batchId][currentStep] = sortedIdVals[offset];

    if (cumLogProbs != nullptr || outputLogProbs != nullptr)
    {
        float lprob = logf(sortedProbs[offset]);
        if (cumLogProbs != nullptr)
        {
            cumLogProbs[batchId] += lprob;
        }
        if (outputLogProbs != nullptr)
        {
            outputLogProbs[sequenceLengths[batchId] * maxBatchSize + batchId] = lprob;
        }
    }
    if (finishedOutput != nullptr && endIds != nullptr)
    {
        if (ids[batchId][currentStep] == endIds[batchId])
        {
            finishedOutput[batchId].setFinishedEOS();
            // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be outputted
        }
        else
        {
            // We don't need to set output finished state as it is assumed to be in non finished state
            sequenceLengths[batchId] += 1;
        }
    }
}

template <typename T, int blockSize>
__global__ void topPSsampling(T const* sortedProbs, TokenIdType const* sortedIdVals, TokenIdType* ids,
    TokenIdType** idsPtrs, SizeType32* sequenceLength, FinishedState const* finishedInput,
    FinishedState* finishedOutput, float* cumLogProbs, float* outputLogProbs, SizeType32 const* beginOffsetBuf,
    SizeType32 const* offsetBuf, SizeType32 vocabSize, hiprandState_t* hiprandState, float const* randomVals,
    float const* topPs, TokenIdType const* endIds, SizeType32 maxBatchSize, bool const* skipDecode,
    SizeType32 const* batchSlots, bool returnAllSelectedTokensFlag, bool const* returnAllSelectedTokensPerSlot,
    SizeType32 maxSeqLen, TokenIdType* outputIdCurrentStep, bool const* skipOutputIdCurrentStep)
{
    /**
     * Each block processes one request row sorted in descending order by probabilities.
     * All threads within block compute running sum of probabilities until one of the threads exceeds the randomly
     * chosen probability threshold. Thread that crossed probaility threshold writes the corresponding token to the
     * output.
     */

    __shared__ float randNumS;
    __shared__ float randNumS2;

    auto const tid = static_cast<SizeType32>(threadIdx.x);
    auto const batchId = static_cast<SizeType32>(blockIdx.x);
    auto const batchSlot = batchSlots[batchId];
    // Skip kernel if this sampling method is not chosen
    FinishedState const finishState = finishedInput != nullptr ? finishedInput[batchSlot] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchSlot]) || (finishState.isSkipDecoding()))
    {
        return;
    }

    // Exit early if sequence has finished
    if (finishState.isFinished())
    {
        if (tid == 0)
        {
            if (finishedOutput != nullptr)
            {
                finishedOutput[batchSlot] = finishState;
            }
        }
        return;
    }

    auto const probThreshold = topPs[batchSlot];
    auto const currentStep = sequenceLength == nullptr ? 0 : sequenceLength[batchSlot];
    auto* outputIdsRequestPtr = idsPtrs == nullptr ? ids + batchSlot * maxSeqLen : idsPtrs[batchSlot];
    auto const returnAllSelectedTokens = returnAllSelectedTokensPerSlot != nullptr
        ? returnAllSelectedTokensPerSlot[batchSlot]
        : returnAllSelectedTokensFlag;
    bool const sampleTokenInSelected = returnAllSelectedTokens && outputIdCurrentStep && hiprandState
        && skipOutputIdCurrentStep && !skipOutputIdCurrentStep[batchSlot];

    // With P in (0.0; 1.0] we draw a random number P' in range (0.0; P]
    // We will sum all probs moving from the largest probability to the smallest and
    // will choose the token which probability makes cumulative probability sum to exceed P'
    if (threadIdx.x == 0)
    {
        // if we want to return all top p indices, we should not do random sampling for probThreshold
        auto const randomNumber = randomVals ? randomVals[batchSlot] : hiprand_uniform(hiprandState + batchSlot);
        randNumS = returnAllSelectedTokens ? probThreshold : randomNumber * probThreshold;
        randNumS2 = sampleTokenInSelected ? hiprand_uniform(hiprandState + batchSlot) * probThreshold : 0.0f;
    }

    // if beginOffsetBuf and offsetBuf of sorting have same value,
    // this means that we have find best one in topPBeamTopKKernel
    // So, we can skip this sampling.
    if (beginOffsetBuf[batchId] == offsetBuf[batchId])
    {
        if (tid == 0)
        {
            auto offset = batchId * vocabSize;
            if (returnAllSelectedTokens)
            {
                outputIdsRequestPtr[currentStep] = sortedIdVals[offset];
            }
            else
            {
                epilogue(batchSlot, currentStep, offset, idsPtrs, sortedIdVals, sortedProbs, cumLogProbs,
                    outputLogProbs, endIds, sequenceLength, finishedOutput, maxBatchSize);
            }
        }
        return;
    }

    typedef hipcub::BlockScan<float, blockSize> BlockScan;
    __shared__ typename BlockScan::TempStorage tempStorage;
    // Initialize running total
    BlockPrefixCallbackOp prefixOp(0);

    __syncthreads();

    auto offset = batchId * vocabSize;
    outputIdsRequestPtr[currentStep] = sortedIdVals[offset];
    auto end = ((vocabSize + blockSize - 1) / blockSize) * blockSize;
    SizeType32 selectedTokenId = 0;
    // Cumulative sum
    float threadOffset = 0;
    SizeType32 count = 0;
    // For sampleTokenInSelected == True
    SizeType32 selectedTokenId2 = 0;
    SizeType32 count2 = 0;
    for (int vi = tid; vi < end; vi += blockSize)
    {
        auto threadProb = (vi < vocabSize) ? static_cast<float>(sortedProbs[offset + vi]) : 0.f;
        BlockScan(tempStorage).InclusiveSum(threadProb, threadOffset, prefixOp);
        count = __syncthreads_count(randNumS <= threadOffset);
        selectedTokenId = vi;
        if (sampleTokenInSelected && count2 == 0)
        {
            count2 = __syncthreads_count(randNumS2 <= threadOffset);
            selectedTokenId2 = vi;
        }
        if (count != 0)
        {
            break;
        }
    }

    selectedTokenId = min(selectedTokenId, vocabSize - 1);

    if (returnAllSelectedTokens)
    {
        __shared__ SizeType32 sharedSelectedTokenId;
        if (sampleTokenInSelected && (threadIdx.x == min(blockDim.x - count2, blockDim.x - 1)))
        {
            selectedTokenId2 = min(selectedTokenId2, vocabSize - 1);
            outputIdCurrentStep[batchSlot] = sortedIdVals[offset + selectedTokenId2];
        }
        if (threadIdx.x == min(blockDim.x - count, blockDim.x - 1))
        {
            sharedSelectedTokenId = selectedTokenId;
        }
        __syncthreads();
        for (int vi = tid; vi <= sharedSelectedTokenId; vi += blockSize)
        {
            outputIdsRequestPtr[vi] = sortedIdVals[offset + vi];
        }
        if (tid == 0 && sharedSelectedTokenId != end - 1)
        {
            outputIdsRequestPtr[sharedSelectedTokenId + 1] = -1; // a boundary to record the end of all selected top Ps.
        }
    }
    else
    {
        // select first thread exceeded the prob threshold or the last thread in case of P=1.0f
        if (threadIdx.x == min(blockDim.x - count, blockDim.x - 1))
        {
            epilogue(batchSlot, currentStep, offset + selectedTokenId, idsPtrs, sortedIdVals, sortedProbs, cumLogProbs,
                outputLogProbs, endIds, sequenceLength, finishedOutput, maxBatchSize);
        }
    }
}

template <typename T>
std::vector<size_t> getTopPWorkspaceSizes(SizeType32 batchSize, SizeType32 vocabSize)
{
    auto const sortedLogProbBufSize = sizeof(T) * batchSize * vocabSize;
    auto const sortedIdValsBufSize = sizeof(TokenIdType) * batchSize * vocabSize;
    auto const topPIdValsSize = sizeof(TokenIdType) * batchSize * vocabSize;
    auto const topPOffsetSize = sizeof(SizeType32) * (batchSize + 1);
    auto const beginTopPOffsetSize = sizeof(SizeType32) * (batchSize + 1);

    size_t cubTempStorageSize;
    tensorrt_llm::common::check_cuda_error(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr,
        cubTempStorageSize, static_cast<T*>(nullptr), static_cast<T*>(nullptr), static_cast<SizeType32*>(nullptr),
        static_cast<SizeType32*>(nullptr), static_cast<SizeType32>(vocabSize * batchSize), batchSize,
        static_cast<SizeType32*>(nullptr), static_cast<SizeType32*>(nullptr),
        0,             // begin_bit
        sizeof(T) * 8, // end_bit = sizeof(KeyT) * 8
        0));           // hipStream_t

    return {cubTempStorageSize, sortedLogProbBufSize, sortedIdValsBufSize, topPIdValsSize, topPOffsetSize,
        beginTopPOffsetSize};
}

template std::vector<size_t> getTopPWorkspaceSizes<float>(SizeType32 batchSize, SizeType32 vocabSize);
template std::vector<size_t> getTopPWorkspaceSizes<half>(SizeType32 batchSize, SizeType32 vocabSize);

[[nodiscard]] std::vector<size_t> getTopPInitWorkspaceSizes(SizeType32 batchSize)
{
    auto const tempTopKsBufSize = batchSize * sizeof(SizeType32);
    auto const tempTopPsBufSize = batchSize * sizeof(float);
    auto const tempTopPDecayBufSize = batchSize * sizeof(float);
    auto const tempTopPMinBufSize = batchSize * sizeof(float);
    auto const tempTopPResetIdsBufSize = batchSize * sizeof(TokenIdType);

    return {tempTopKsBufSize, tempTopPsBufSize, tempTopPDecayBufSize, tempTopPMinBufSize, tempTopPResetIdsBufSize};
}

template <typename T>
size_t getTopPWorkspaceSize(SizeType32 batchSize, SizeType32 vocabSizePadded)
{
    auto const workspaceSizes = getTopPWorkspaceSizes<T>(batchSize, vocabSizePadded);
    auto const initWorkspaceSizes = getTopPInitWorkspaceSizes(batchSize);
    return std::max(tensorrt_llm::common::calcAlignedSize(workspaceSizes, 256),
        tensorrt_llm::common::calcAlignedSize(initWorkspaceSizes, 256));
}

template size_t getTopPWorkspaceSize<float>(SizeType32 batchSize, SizeType32 vocabSizePadded);
template size_t getTopPWorkspaceSize<half>(SizeType32 batchSize, SizeType32 vocabSizePadded);

template <typename T>
void invokeBatchTopPSampling(TopPSamplingKernelParams<T> const& params, hipStream_t stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    params.checkParams();

    auto const workspaceSizes = getTopPWorkspaceSizes<T>(params.batchSize, params.vocabSizePadded);

    std::vector<void*> alignedPointers;
    calcAlignedPointers(alignedPointers, params.workspace, workspaceSizes);

    auto cubTempStorage = static_cast<void*>(alignedPointers[0]);
    auto sortedProbs = static_cast<T*>(alignedPointers[1]);
    auto sortedIdVals = static_cast<TokenIdType*>(alignedPointers[2]);
    auto idVals = static_cast<TokenIdType*>(alignedPointers[3]);
    auto offsetBuf = static_cast<SizeType32*>(alignedPointers[4]);
    auto beginOffsetBuf = static_cast<SizeType32*>(alignedPointers[5]);

    invokeTopPInitialize(idVals, offsetBuf, beginOffsetBuf, params.batchSize, params.vocabSizePadded, stream);
    sync_check_cuda_error(stream);

    SizeType32 constexpr BLOCK_SIZE = 256;
    // Performs Top K=1 search.
    // If the most probable token exceeds P, we skip sorting by setting beginOffsetBuf[bi] = offsetBuf[bi]
    topPBeamTopKKernel<T, BLOCK_SIZE><<<params.batchSize, BLOCK_SIZE, 0, stream>>>(params.probs, sortedIdVals,
        sortedProbs, params.finishedInput, params.vocabSizePadded, offsetBuf, beginOffsetBuf, params.topPs,
        params.skipDecode, params.batchSlots);
    sync_check_cuda_error(stream);

    // Sort tokens by probability in descending order
    auto cubWorkspaceSize = workspaceSizes[0];
    check_cuda_error(hipcub::DeviceSegmentedRadixSort::SortPairsDescending(cubTempStorage, cubWorkspaceSize, params.probs,
        sortedProbs, idVals, sortedIdVals, params.vocabSizePadded * params.batchSize, params.batchSize, beginOffsetBuf,
        offsetBuf + 1,
        0,                                      // begin_bit
        static_cast<SizeType32>(sizeof(T) * 8), // end_bit = sizeof(KeyT) * 8
        stream));                               // hipStream_t

    SizeType32 constexpr SAMPLING_BLOCK_SIZE = 256;
    dim3 grid(params.batchSize);
    // Sample with Top P given sorted tokens
    topPSsampling<T, SAMPLING_BLOCK_SIZE><<<grid, SAMPLING_BLOCK_SIZE, 0, stream>>>(sortedProbs, sortedIdVals,
        params.outputIds, params.outputIdsPtrs, params.sequenceLength, params.finishedInput, params.finishedOutput,
        params.cumLogProbs, params.outputLogProbs, beginOffsetBuf, offsetBuf + 1, params.vocabSizePadded,
        params.hiprandState, params.randomVals, params.topPs, params.endIds, params.maxBatchSize, params.skipDecode,
        params.batchSlots, params.returnAllSelectedTokens, params.returnAllSelectedTokensPerSlot, params.maxSeqLen,
        params.outputIdCurrentStep, params.skipOutputIdCurrentStep);
    sync_check_cuda_error(stream);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template void invokeBatchTopPSampling(TopPSamplingKernelParams<float> const& params, hipStream_t stream);

template void invokeBatchTopPSampling(TopPSamplingKernelParams<half> const& params, hipStream_t stream);

__global__ void computeToppDecay(float* runtimeTopP, float const* runtimeInitialTopP, TokenIdType const** outputIds,
    float const* topPDecay, float const* topPMin, TokenIdType const* topPResetIds, SizeType32 const* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 localBatchSize)
{
    auto const idx = static_cast<SizeType32>(blockDim.x * blockIdx.x + threadIdx.x);
    if (idx >= localBatchSize)
    {
        return;
    }
    auto const batchSlot = batchSlots[idx];
    auto const currentStep{sequenceLengths[batchSlot]};
    if (outputIds[batchSlot][currentStep] == topPResetIds[batchSlot])
    {
        runtimeTopP[batchSlot] = runtimeInitialTopP[batchSlot];
    }
    else
    {
        runtimeTopP[batchSlot] = max(runtimeTopP[batchSlot] * topPDecay[batchSlot], topPMin[batchSlot]);
    }
}

void invokeComputeToppDecay(float* runtimeTopP, float const* runtimeInitialTopP, TokenIdType const** outputIds,
    float const* topPDecay, float const* topPMin, TokenIdType const* topPResetIds, SizeType32 const* sequenceLengths,
    SizeType32 const* batchSlots, SizeType32 localBatchSize, hipStream_t stream)
{
    dim3 block(std::min(localBatchSize, 512));
    dim3 grid((localBatchSize + block.x - 1) / block.x);
    computeToppDecay<<<grid, block, 0, stream>>>(runtimeTopP, runtimeInitialTopP, outputIds, topPDecay, topPMin,
        topPResetIds, sequenceLengths, batchSlots, localBatchSize);
}

__global__ void setTopPRuntimeArgs(SizeType32 batchSize, SizeType32 const* batchSlots,
    ScatterDecodingParamEntry<SizeType32> topK, ScatterDecodingParamEntry<float> topP, bool* skipDecode,
    float* initialTopPBuf)
{
    auto index = static_cast<SizeType32>(blockIdx.x * blockDim.x + threadIdx.x);
    for (SizeType32 bi = index; bi < batchSize; bi += static_cast<SizeType32>(gridDim.x * blockDim.x))
    {
        setupTopKTopPRuntimeArgOne(bi, topK, topP, batchSlots, nullptr, skipDecode, initialTopPBuf);
    }
}

void invokeSetTopPRuntimeArgs(SizeType32 batchSize, ScatterDecodingParamEntry<SizeType32> topK,
    ScatterDecodingParamEntry<float> topP, bool* skipDecodePtr, float* initialTopPPtr, SizeType32 const* batchSlotsPtr,
    bool onDevice, hipStream_t stream)
{
    if (onDevice)
    {
        dim3 block(std::min(static_cast<uint32_t>(batchSize), 256u));
        dim3 grid(divUp(static_cast<uint32_t>(batchSize), block.x));
        setTopPRuntimeArgs<<<grid, block, 0, stream>>>(
            batchSize, batchSlotsPtr, topK, topP, skipDecodePtr, initialTopPPtr);
    }
    else
    {
        for (int bi = 0; bi < batchSize; ++bi)
        {
            setupTopKTopPRuntimeArgOne(bi, topK, topP, batchSlotsPtr, nullptr, skipDecodePtr, nullptr);
        }
    }
}

} // namespace tensorrt_llm::kernels
