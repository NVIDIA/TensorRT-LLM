#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/kernels/preQuantScaleKernel.h"

namespace tensorrt_llm
{
namespace kernels
{
namespace
{
template <typename T>
struct Vec2Type;

template <>
struct Vec2Type<half>
{
    using type = half2;
};
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800) && defined(ENABLE_BF16))
template <>
struct Vec2Type<__hip_bfloat16>
{
    using type = __hip_bfloat162;
};
#endif
}; // namespace

template <typename T_in, typename T_out, int kProcessRows, typename AccessType>
__global__ void apply_per_channel_scale(
    T_out* smoothed_act, T_in const* act, T_in const* per_channel_scale, int rows, int cols)
{
    static constexpr int kElems = sizeof(AccessType) / sizeof(T_in);
    T_in scale[kElems], act_vec[kElems];
    int col_offset = blockIdx.y * blockDim.x + threadIdx.x;
    int row_offset = blockIdx.x;
    if (col_offset * kElems >= cols || row_offset * kProcessRows >= rows)
        return;
    act += row_offset * kProcessRows * cols;
    smoothed_act += row_offset * kProcessRows * cols;
    *reinterpret_cast<AccessType*>(scale) = reinterpret_cast<AccessType const*>(per_channel_scale)[col_offset];
#pragma unroll
    for (int i = 0; i < kProcessRows; ++i)
    {
        *reinterpret_cast<AccessType*>(act_vec) = reinterpret_cast<AccessType const*>(act + i * cols)[col_offset];
        if constexpr ((std::is_same_v<T_in, half>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800) && defined(ENABLE_BF16))
                          || std::is_same_v<T_in, __hip_bfloat16>
#endif
                          ) &&(kElems % 2 == 0))
        {
            using Vec2 = typename Vec2Type<T_in>::type;
#pragma unroll
            for (int j = 0; j < kElems; j += 2)
            {
                *reinterpret_cast<Vec2*>(act_vec + j)
                    = __hmul2(*reinterpret_cast<Vec2*>(act_vec + j), *reinterpret_cast<Vec2*>(scale + j));
            }
        }
        else
        {
#pragma unroll
            for (int j = 0; j < kElems; ++j)
            {
                act_vec[j] = static_cast<T_in>(static_cast<float>(act_vec[j]) * static_cast<float>(scale[j]));
            }
        }
        if constexpr (std::is_same_v<T_in, T_out>)
        {
            reinterpret_cast<AccessType*>(smoothed_act + i * cols)[col_offset]
                = *reinterpret_cast<AccessType*>(act_vec);
        }
        else
        {
#pragma unroll
            for (int j = 0; j < kElems; ++j)
            {
                (smoothed_act + i * cols)[col_offset * kElems + j] = static_cast<T_out>(act_vec[j]);
            }
        }
    }
}

template <typename T_in, typename T_out, int kProcessRows, typename AccessType = float4>
void apply_per_channel_scale_kernel_launcher_(
    T_out* smoothed_act, T_in const* act, T_in const* per_channel_scale, int rows, int cols, hipStream_t stream = 0)
{
    static constexpr int kElems = sizeof(AccessType) / sizeof(T_in);
    dim3 block(128);
    dim3 grid((rows + kProcessRows - 1) / kProcessRows, (cols / kElems + block.x - 1) / block.x);
    apply_per_channel_scale<T_in, T_out, kProcessRows, AccessType>
        <<<grid, block, 0, stream>>>(smoothed_act, act, per_channel_scale, rows, cols);
}

template <typename T_in, typename T_out>
void apply_per_channel_scale_kernel_launcher(
    T_out* smoothed_act, T_in const* act, T_in const* per_channel_scale, int rows, int cols, hipStream_t stream)
{
    uint64_t elems = static_cast<uint64_t>(rows) * static_cast<uint64_t>(cols);
    if (elems < 2048 * 2048)
    {
        apply_per_channel_scale_kernel_launcher_<T_in, T_out, 1, float4>(
            smoothed_act, act, per_channel_scale, rows, cols, stream);
    }
    else if (elems < 4096 * 4096)
    {
        apply_per_channel_scale_kernel_launcher_<T_in, T_out, 4, float4>(
            smoothed_act, act, per_channel_scale, rows, cols, stream);
    }
    else if (elems < 8192 * 8192)
    {
        apply_per_channel_scale_kernel_launcher_<T_in, T_out, 8, float4>(
            smoothed_act, act, per_channel_scale, rows, cols, stream);
    }
    else
    {
        apply_per_channel_scale_kernel_launcher_<T_in, T_out, 16, float4>(
            smoothed_act, act, per_channel_scale, rows, cols, stream);
    }
}

#define INSTANTIATE_PREQUANT_SCALE(T_in, T_out)                                                                        \
    template void apply_per_channel_scale_kernel_launcher<T_in, T_out>(                                                \
        T_out * smoothed_act, const T_in* act, const T_in* per_channel_scale, int rows, int cols, hipStream_t stream)

INSTANTIATE_PREQUANT_SCALE(half, half);
#if defined(ENABLE_FP8)
INSTANTIATE_PREQUANT_SCALE(half, __hip_fp8_e4m3_fnuz);
#endif

#if defined(ENABLE_BF16)
INSTANTIATE_PREQUANT_SCALE(__hip_bfloat16, __hip_bfloat16);
#if defined(ENABLE_FP8)
INSTANTIATE_PREQUANT_SCALE(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

template <typename T_in, typename T_out, int kProcessRows, typename AccessType>
__global__ void apply_per_expert_scale(T_out* smoothed_act, T_in const* act, T_in const* per_expert_scale,
    int const* permuted_token_selected_experts, int64_t const* num_valid_tokens_ptr, int rows, int cols)
{
    static constexpr int kElems = sizeof(AccessType) / sizeof(T_in);
    T_in act_vec[kElems];
    int col_offset = blockIdx.x * blockDim.x + threadIdx.x;
    int row_offset = blockIdx.y;
    int expert_idx = permuted_token_selected_experts[row_offset];
    T_in scale = per_expert_scale[expert_idx];
    if (col_offset * kElems >= cols || row_offset * kProcessRows >= rows)
        return;
    if (num_valid_tokens_ptr && (row_offset * kProcessRows >= *num_valid_tokens_ptr))
        return;
    act += row_offset * kProcessRows * cols;
    smoothed_act += row_offset * kProcessRows * cols;
#pragma unroll
    for (int i = 0; i < kProcessRows; ++i)
    {
        *reinterpret_cast<AccessType*>(act_vec) = reinterpret_cast<AccessType const*>(act + i * cols)[col_offset];
        if constexpr ((std::is_same_v<T_in, half>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800) && defined(ENABLE_BF16))
                          || std::is_same_v<T_in, __hip_bfloat16>
#endif
                          ) &&(kElems % 2 == 0))
        {
            using Vec2 = typename Vec2Type<T_in>::type;
#pragma unroll
            for (int j = 0; j < kElems; j += 2)
            {
                if constexpr (std::is_same_v<T_in, half>)
                {
                    *reinterpret_cast<Vec2*>(act_vec + j)
                        = __hmul2(*reinterpret_cast<Vec2*>(act_vec + j), __half2half2(scale));
                }
                else
                {
                    *reinterpret_cast<Vec2*>(act_vec + j)
                        = __hmul2(*reinterpret_cast<Vec2*>(act_vec + j), __bfloat162bfloat162(scale));
                }
            }
        }
        else
        {
#pragma unroll
            for (int j = 0; j < kElems; ++j)
            {
                act_vec[j] = static_cast<T_in>(static_cast<float>(act_vec[j]) * static_cast<float>(scale));
            }
        }
        if constexpr (std::is_same_v<T_in, T_out>)
        {
            reinterpret_cast<AccessType*>(smoothed_act + i * cols)[col_offset]
                = *reinterpret_cast<AccessType*>(act_vec);
        }
        else
        {
#pragma unroll
            for (int j = 0; j < kElems; ++j)
            {
                (smoothed_act + i * cols)[col_offset * kElems + j] = static_cast<T_out>(act_vec[j]);
            }
        }
    }
}

template <typename T_in, typename T_out, int kProcessRows, typename AccessType = float4>
void apply_per_expert_scale_kernel_launcher_(T_out* smoothed_act, T_in const* act, T_in const* per_expert_scale,
    int const* permuted_token_selected_experts, int64_t const* num_valid_tokens_ptr, int rows, int cols,
    hipStream_t stream = 0)
{
    static constexpr int kElems = sizeof(AccessType) / sizeof(T_in);
    dim3 block(128);
    dim3 grid((cols / kElems + block.x - 1) / block.x, (rows + kProcessRows - 1) / kProcessRows);
    apply_per_expert_scale<T_in, T_out, kProcessRows, AccessType><<<grid, block, 0, stream>>>(
        smoothed_act, act, per_expert_scale, permuted_token_selected_experts, num_valid_tokens_ptr, rows, cols);
}

template <typename T_in, typename T_out>
void apply_per_expert_scale_kernel_launcher(T_out* smoothed_act, T_in const* act, T_in const* per_expert_scale,
    int const* permuted_token_selected_experts, int64_t const* num_valid_tokens_ptr, int rows, int cols,
    hipStream_t stream)
{
    int elems = rows * cols;
    if (elems < 2048 * 2048)
    {
        apply_per_expert_scale_kernel_launcher_<T_in, T_out, 1, float4>(smoothed_act, act, per_expert_scale,
            permuted_token_selected_experts, num_valid_tokens_ptr, rows, cols, stream);
    }
    else if (elems < 4096 * 4096)
    {
        apply_per_expert_scale_kernel_launcher_<T_in, T_out, 4, float4>(smoothed_act, act, per_expert_scale,
            permuted_token_selected_experts, num_valid_tokens_ptr, rows, cols, stream);
    }
    else if (elems < 8192 * 8192)
    {
        apply_per_expert_scale_kernel_launcher_<T_in, T_out, 8, float4>(smoothed_act, act, per_expert_scale,
            permuted_token_selected_experts, num_valid_tokens_ptr, rows, cols, stream);
    }
    else
    {
        apply_per_expert_scale_kernel_launcher_<T_in, T_out, 16, float4>(smoothed_act, act, per_expert_scale,
            permuted_token_selected_experts, num_valid_tokens_ptr, rows, cols, stream);
    }
}

#define INSTANTIATE_PEREXPERT_SCALE(T_in, T_out)                                                                       \
    template void apply_per_expert_scale_kernel_launcher<T_in, T_out>(T_out * smoothed_act, T_in const* act,           \
        T_in const* per_expert_scale, int const* permuted_token_selected_experts, int64_t const* num_valid_tokens_ptr, \
        int rows, int cols, hipStream_t stream)

INSTANTIATE_PEREXPERT_SCALE(half, half);
#if defined(ENABLE_FP8)
INSTANTIATE_PEREXPERT_SCALE(half, __hip_fp8_e4m3_fnuz);
#endif

#if defined(ENABLE_BF16)
INSTANTIATE_PEREXPERT_SCALE(__hip_bfloat16, __hip_bfloat16);
#if defined(ENABLE_FP8)
INSTANTIATE_PEREXPERT_SCALE(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

} // namespace kernels
} // namespace tensorrt_llm
