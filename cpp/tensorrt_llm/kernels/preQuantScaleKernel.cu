#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2022-2024 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/kernels/preQuantScaleKernel.h"

namespace tensorrt_llm
{
namespace kernels
{
namespace
{
template <typename T>
struct Vec2Type;

template <>
struct Vec2Type<half>
{
    using type = half2;
};
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800) && defined(ENABLE_BF16))
template <>
struct Vec2Type<__hip_bfloat16>
{
    using type = __hip_bfloat162;
};
#endif
}; // namespace

template <typename T_in, typename T_out, int kProcessRows, typename AccessType>
__global__ void apply_per_channel_scale(T_out* smoothed_act, T_in const* act, T_in const* per_channel_scale, int rows,
    int cols, int64_t const* num_valid_tokens_ptr)
{
    static constexpr int kElems = sizeof(AccessType) / sizeof(T_in);
    T_in scale[kElems], act_vec[kElems];
    int col_offset = blockIdx.y * blockDim.x + threadIdx.x;
    int row_offset = blockIdx.x;
    if (col_offset * kElems >= cols || row_offset * kProcessRows >= rows)
        return;
    if (num_valid_tokens_ptr && (row_offset * kProcessRows >= *num_valid_tokens_ptr))
        return;
    act += row_offset * kProcessRows * cols;
    smoothed_act += row_offset * kProcessRows * cols;
    *reinterpret_cast<AccessType*>(scale) = reinterpret_cast<AccessType const*>(per_channel_scale)[col_offset];
#pragma unroll
    for (int i = 0; i < kProcessRows; ++i)
    {
        *reinterpret_cast<AccessType*>(act_vec) = reinterpret_cast<AccessType const*>(act + i * cols)[col_offset];
        if constexpr ((std::is_same_v<T_in, half>
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800) && defined(ENABLE_BF16))
                          || std::is_same_v<T_in, __hip_bfloat16>
#endif
                          ) &&(kElems % 2 == 0))
        {
            using Vec2 = typename Vec2Type<T_in>::type;
#pragma unroll
            for (int j = 0; j < kElems; j += 2)
            {
                *reinterpret_cast<Vec2*>(act_vec + j)
                    = __hmul2(*reinterpret_cast<Vec2*>(act_vec + j), *reinterpret_cast<Vec2*>(scale + j));
            }
        }
        else
        {
#pragma unroll
            for (int j = 0; j < kElems; ++j)
            {
                act_vec[j] = static_cast<T_in>(static_cast<float>(act_vec[j]) * static_cast<float>(scale[j]));
            }
        }
        if constexpr (std::is_same_v<T_in, T_out>)
        {
            reinterpret_cast<AccessType*>(smoothed_act + i * cols)[col_offset]
                = *reinterpret_cast<AccessType*>(act_vec);
        }
        else
        {
#pragma unroll
            for (int j = 0; j < kElems; ++j)
            {
                (smoothed_act + i * cols)[col_offset * kElems + j] = static_cast<T_out>(act_vec[j]);
            }
        }
    }
}

template <typename T_in, typename T_out, int kProcessRows, typename AccessType = float4>
void apply_per_channel_scale_kernel_launcher_(T_out* smoothed_act, T_in const* act, T_in const* per_channel_scale,
    int rows, int cols, int64_t const* num_valid_tokens_ptr = nullptr, hipStream_t stream = 0)
{
    static constexpr int kElems = sizeof(AccessType) / sizeof(T_in);
    dim3 block(128);
    dim3 grid((rows + kProcessRows - 1) / kProcessRows, (cols / kElems + block.x - 1) / block.x);
    apply_per_channel_scale<T_in, T_out, kProcessRows, AccessType>
        <<<grid, block, 0, stream>>>(smoothed_act, act, per_channel_scale, rows, cols, num_valid_tokens_ptr);
}

template <typename T_in, typename T_out>
void apply_per_channel_scale_kernel_launcher(T_out* smoothed_act, T_in const* act, T_in const* per_channel_scale,
    int rows, int cols, int64_t const* num_valid_tokens_ptr, hipStream_t stream)
{
    uint64_t elems = static_cast<uint64_t>(rows) * static_cast<uint64_t>(cols);
    if (elems < 2048 * 2048)
    {
        apply_per_channel_scale_kernel_launcher_<T_in, T_out, 1, float4>(
            smoothed_act, act, per_channel_scale, rows, cols, num_valid_tokens_ptr, stream);
    }
    else if (elems < 4096 * 4096)
    {
        apply_per_channel_scale_kernel_launcher_<T_in, T_out, 4, float4>(
            smoothed_act, act, per_channel_scale, rows, cols, num_valid_tokens_ptr, stream);
    }
    else if (elems < 8192 * 8192)
    {
        apply_per_channel_scale_kernel_launcher_<T_in, T_out, 8, float4>(
            smoothed_act, act, per_channel_scale, rows, cols, num_valid_tokens_ptr, stream);
    }
    else
    {
        apply_per_channel_scale_kernel_launcher_<T_in, T_out, 16, float4>(
            smoothed_act, act, per_channel_scale, rows, cols, num_valid_tokens_ptr, stream);
    }
}

#define INSTANTIATE_PREQUANT_SCALE(T_in, T_out)                                                                        \
    template void apply_per_channel_scale_kernel_launcher<T_in, T_out>(T_out * smoothed_act, const T_in* act,          \
        const T_in* per_channel_scale, int rows, int cols, int64_t const* num_valid_tokens_ptr, hipStream_t stream)

INSTANTIATE_PREQUANT_SCALE(half, half);
#if defined(ENABLE_FP8)
INSTANTIATE_PREQUANT_SCALE(half, __hip_fp8_e4m3_fnuz);
#endif

#if defined(ENABLE_BF16)
INSTANTIATE_PREQUANT_SCALE(__hip_bfloat16, __hip_bfloat16);
#if defined(ENABLE_FP8)
INSTANTIATE_PREQUANT_SCALE(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

} // namespace kernels
} // namespace tensorrt_llm
