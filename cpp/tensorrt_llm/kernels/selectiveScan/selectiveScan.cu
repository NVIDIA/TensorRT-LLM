#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <hip/hip_runtime_api.h>

#include <cooperative_groups/memcpy_async.h>
#include <cuda/pipeline>

#include <hip/hip_runtime.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>

#ifdef ENABLE_FP8
#include <hip/hip_fp8.h>
#endif

#include "selectiveScan.h"

#include "CudaType.h"
#include "bmmchunk.h"
#include "chunkcumsum.h"
#include "chunkscan.h"
#include "chunkstate.h"
#include "statepassing.h"

namespace tensorrt_llm
{
namespace kernels
{

__device__ float toFloat(float f)
{
    return f;
}

__device__ float toFloat(__half h)
{
    return __half2float(h);
}
#ifdef ENABLE_BF16
__device__ float toFloat(__hip_bfloat16 val)
{
    return __bfloat162float(val);
}
#endif

__device__ void convertAndStore(float* output, float input)
{
    *output = input;
}

__device__ void convertAndStore(__half* output, float input)
{
    *output = __float2half(input);
}
#ifdef ENABLE_BF16
__device__ void convertAndStore(__hip_bfloat16* output, float input)
{
    *output = __float2bfloat16(input);
}
#endif

#pragma nv_diag_suppress static_var_with_dynamic_init

template <typename input_t, typename weight_t, int DSTATE = 16, int CHANNELS_PER_BLOCK = 128, int STAGES = 12,
    int SEQ_UNROLL = 6>
__launch_bounds__(256, 1) __global__ void selective_scan_loop_kernel(SSMParamsBase params)
{
    input_t* output = reinterpret_cast<input_t*>(params.out_ptr);
    input_t* state = reinterpret_cast<input_t*>(params.x_ptr);
    input_t* x = reinterpret_cast<input_t*>(params.u_ptr);
    input_t* dt = reinterpret_cast<input_t*>(params.delta_ptr);
    weight_t* A = reinterpret_cast<weight_t*>(params.A_ptr);
    input_t* B = reinterpret_cast<input_t*>(params.BC_ptr);
    input_t* C = reinterpret_cast<input_t*>(params.BC_ptr);
    weight_t* D = reinterpret_cast<weight_t*>(params.D_ptr);
    input_t* z = reinterpret_cast<input_t*>(params.z_ptr);
    weight_t* dt_bias = reinterpret_cast<weight_t*>(params.delta_bias_ptr);
    bool dt_softplus = params.delta_softplus;
    int num_channels = params.dim;

    __shared__ cuda::pipeline_shared_state<cuda::thread_scope::thread_scope_block, STAGES / SEQ_UNROLL> pipeline_state;
    auto block = cooperative_groups::this_thread_block();

    __shared__ __align__(16) input_t sh_B[STAGES][DSTATE];
    __shared__ __align__(16) input_t sh_C[STAGES][DSTATE];

    __shared__ __align__(128) input_t sh_dt[STAGES][CHANNELS_PER_BLOCK];
    __shared__ input_t sh_x[STAGES][CHANNELS_PER_BLOCK];
    __shared__ input_t sh_z[STAGES][CHANNELS_PER_BLOCK];

    int const channel = blockIdx.x * blockDim.x + threadIdx.x;
    int const sample = blockIdx.y; // batch id

    int const slot_idx = params.slot_mapping_ptr == nullptr ? sample : params.slot_mapping_ptr[sample];
    int const bc_cols = DSTATE * 2 + params.dt_rank;
    int const b_offset = params.dt_rank;
    int const c_offset = params.dt_rank + DSTATE;

    int num_tokens;
    int start_token_idx;
    if (params.remove_padding)
    {
        start_token_idx = sample == 0 ? 0 : params.last_token_ids_ptr[sample - 1];
        int end_token_idx = params.last_token_ids_ptr[sample];
        num_tokens = end_token_idx - start_token_idx;
    }
    else
    {
        start_token_idx = sample * params.max_seqlen;
        num_tokens = params.last_token_ids_ptr[sample];
    }
    int const seq_loops = (num_tokens + SEQ_UNROLL - 1) / SEQ_UNROLL;

    int const input_matrix_row_id = start_token_idx;

    if (threadIdx.y == 1)
    {
        cuda::pipeline pipeline = cuda::make_pipeline(block, &pipeline_state, cuda::pipeline_role::producer);

        int stage = 0;
        for (int si = 0; si < seq_loops; si++)
        {

            pipeline.producer_acquire();

#pragma unroll
            for (int token_id = si * SEQ_UNROLL; token_id < num_tokens && token_id < (si + 1) * SEQ_UNROLL; token_id++)
            {

                input_t* my_B = &B[(input_matrix_row_id + token_id) * bc_cols + b_offset];
                input_t* my_C = &C[(input_matrix_row_id + token_id) * bc_cols + c_offset];

                int block_channel_per_token = blockIdx.x * blockDim.x;
                int block_channel
                    = input_matrix_row_id * num_channels + token_id * num_channels + block_channel_per_token;

                if (threadIdx.x < DSTATE)
                    cuda::memcpy_async(&sh_B[stage][threadIdx.x], &my_B[threadIdx.x], sizeof(input_t), pipeline);
                else if (threadIdx.x >= 32 && threadIdx.x < 32 + DSTATE)
                    cuda::memcpy_async(
                        &sh_C[stage][threadIdx.x - 32], &my_C[threadIdx.x - 32], sizeof(input_t), pipeline);
                if (sizeof(input_t) == 4)
                {
                    cuda::memcpy_async(&sh_dt[stage][threadIdx.x],
                        &dt[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                        pipeline);
                    cuda::memcpy_async(&sh_x[stage][threadIdx.x],
                        &x[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                        pipeline);
                    if (z)
                        cuda::memcpy_async(&sh_z[stage][threadIdx.x],
                            &z[input_matrix_row_id * num_channels + token_id * num_channels + channel], sizeof(input_t),
                            pipeline);
                }
                else
                {
                    // sh_dt[stage][threadIdx.x] = dt[block_channel + threadIdx.x];
                    if (threadIdx.x < 32)
                    {
                        int tid = threadIdx.x;
                        float2* block_dt = (float2*) &dt[block_channel];
                        cuda::memcpy_async((float2*) &sh_dt[stage][tid * 4], &block_dt[tid], sizeof(float2), pipeline);
                    }
                    // sh_x[stage][threadIdx.x] = x[block_channel + threadIdx.x];
                    else if (threadIdx.x < 64)
                    {
                        int tid = threadIdx.x - 32;
                        float2* block_x = (float2*) &x[block_channel];
                        cuda::memcpy_async((float2*) &sh_x[stage][tid * 4], &block_x[tid], sizeof(float2), pipeline);
                    }
                    // sh_z[stage][threadIdx.x] = z[block_channel + threadIdx.x];
                    else if (threadIdx.x < 96)
                    {
                        int tid = threadIdx.x - 64;
                        if (z)
                        {
                            float2* block_z = (float2*) &z[block_channel];
                            cuda::memcpy_async(
                                (float2*) &sh_z[stage][tid * 4], &block_z[tid], sizeof(float2), pipeline);
                        }
                    }
                    else
                    {
                    }
                }

                stage++;
                if (stage >= STAGES)
                    stage = 0;
            }
            pipeline.producer_commit();
        }
    }
    else
    {

        // Compute warps
        // Load state and A matrix into registers
        float state_reg[DSTATE];
        float A_reg[DSTATE];
        for (int i = 0; i < DSTATE; i++)
        {
            state_reg[i] = 0.f;
            A_reg[i] = toFloat(A[i * num_channels + channel]);
        }
        float dt_bias_reg = dt_bias[channel];
        float D_reg = D ? D[channel] : 0.f;

        cuda::pipeline pipeline = cuda::make_pipeline(block, &pipeline_state, cuda::pipeline_role::consumer);
        int stage = 0;
        for (int si = 0; si < seq_loops; si++)
        {

            pipeline.consumer_wait();

#pragma unroll
            for (int token_id = si * SEQ_UNROLL; token_id < num_tokens && token_id < (si + 1) * SEQ_UNROLL; token_id++)
            {

                float dt_b = toFloat(sh_dt[stage][threadIdx.x]) + dt_bias_reg;
                float dt_b_sp;
                if (dt_softplus)
                {
                    dt_b_sp = dt_b <= 20.f ? __logf(1.f + __expf(dt_b)) : dt_b; // softplus
                }
                float my_x = toFloat(sh_x[stage][threadIdx.x]);
                float Dx = my_x * D_reg;
                float dtx = dt_b_sp * my_x;
                float my_z = z ? toFloat(sh_z[stage][threadIdx.x]) : 0.f;

                float out = Dx;

                if (sizeof(input_t) == 4)
                {
                    float4* B4 = (float4*) &sh_B[stage][0];
                    float4* C4 = (float4*) &sh_C[stage][0];
#pragma unroll
                    for (int i = 0; i < DSTATE / 4; i++)
                    {

                        float4 Bi4 = B4[i];
                        float4 Ci4 = C4[i];

                        float* Bi = (float*) &Bi4;
                        float* Ci = (float*) &Ci4;

#pragma unroll
                        for (int j = 0; j < 4; j++)
                        {
                            float dtA = A_reg[i * 4 + j] * dt_b_sp;
                            float dA = __expf(dtA);
                            float sdA = state_reg[i * 4 + j] * dA;
                            float dBx = Bi[j] * dtx;
                            float newState = sdA + dBx;
                            state_reg[i * 4 + j] = newState;
                            out += newState * Ci[j];
                        }
                    }
                }
                else
                {
                    float4* B8 = (float4*) &sh_B[stage][0];
                    float4* C8 = (float4*) &sh_C[stage][0];
#pragma unroll
                    for (int i = 0; i < DSTATE / 8; i++)
                    {
                        input_t* Bi = (input_t*) (&B8[i]);
                        input_t* Ci = (input_t*) (&C8[i]);
#pragma unroll
                        for (int j = 0; j < 8; j++)
                        {
                            float dtA = A_reg[i * 8 + j] * dt_b_sp;
                            float dA = __expf(dtA);
                            float sdA = state_reg[i * 8 + j] * dA;
                            float dBx = toFloat(Bi[j]) * dtx;
                            float newState = sdA + dBx;
                            state_reg[i * 8 + j] = newState;
                            out += newState * toFloat(Ci[j]);
                        }
                    }
                }

                if (z)
                {
                    float enz = __expf(0.f - my_z);
                    enz += 1.0;
                    float sig_z = __fdividef(1.f, enz);
                    float silu_z = my_z * sig_z;
                    out *= silu_z;
                }
                input_t* my_output = &output[input_matrix_row_id * num_channels + token_id * num_channels];
                convertAndStore(&my_output[channel], out);

                stage++;
                if (stage >= STAGES)
                    stage = 0;
            }
            pipeline.consumer_release();
        }
        // Write the new state back out to the cache
        for (int i = 0; i < DSTATE; i++)
        {
            input_t* my_state = &state[slot_idx * num_channels * DSTATE];
            int offset = i * num_channels + channel;
            convertAndStore(&my_state[offset], state_reg[i]);
        }
    }
}

template <typename input_t, typename weight_t>
void invokeSelectiveScan(SSMParamsBase& params, hipStream_t stream)
{
    int samples = params.batch;
    int channels = params.dim;

    TLLM_CHECK(params.dstate == 16);

    int const threads = 128;
    int const blocks = (channels + threads - 1) / threads;
    dim3 block(threads, 2);
    dim3 grid(blocks, samples);
    TLLM_CHECK((channels % block.x) == 0);
    selective_scan_loop_kernel<input_t, weight_t><<<grid, block, 0, stream>>>(params);
}

template <typename input_t, typename weight_t>
void invokeChunkScan(SSMParamsBase& params, hipStream_t stream, tensorrt_llm::common::CUDADriverWrapper* driver)
{
    int B = params.batch;
    int L = params.max_seqlen;
    int H = params.nheads;
    int P = params.dim / H;
    int G = params.ngroups;
    int N = params.dstate;
    int Q = params.chunk_size;

    int numTokens = params.num_tokens;

    bool dtsp = params.delta_softplus;

    bool hopper = tensorrt_llm::common::getSMVersion() >= 90 && tensorrt_llm::common::getSMVersion() < 100;

    CudaType tp, wt;

    if (std::is_same_v<input_t, half>)
        tp = CT_FP16;
    else if (std::is_same_v<input_t, __hip_bfloat16>)
        tp = CT_BF16;
    else
        return;

    if (std::is_same_v<weight_t, float>)
        wt = CT_FP32;
    else if (std::is_same_v<weight_t, input_t>)
        wt = tp;
    else
        return;

    dim3 bds[5], tds[5];
    int shms[5], useTmas[5];
    CUtensorMap descs_host[8];

    ChunkCumsumKernelFunc chunk_cumsum
        = getChunkCumsumKernel(B, L, H, P, G, N, Q, numTokens, &bds[0], &tds[0], &shms[0], tp, wt);
    ChunkStateKernelFunc chunk_state = getChunkStateKernel(
        B, L, H, P, G, N, Q, numTokens, hopper, driver, &bds[1], &tds[1], &shms[1], &useTmas[1], &descs_host[0], tp);
    StatePassingKernelFunc state_passing
        = getStatePassingKernel(B, L, H, P, G, N, Q, numTokens, &bds[2], &tds[2], &shms[2], tp);
    BmmChunkKernelFunc bmm_chunk = getBmmChunkKernel(
        B, L, H, P, G, N, Q, numTokens, hopper, driver, &bds[3], &tds[3], &shms[3], &useTmas[3], &descs_host[2], tp);
    ChunkScanKernelFunc chunk_scan = getChunkScanKernel(B, L, H, P, G, N, Q, numTokens, hopper, driver, &bds[4],
        &tds[4], &shms[4], &useTmas[4], &descs_host[4], tp, wt);

    void* mxY = params.out_ptr;
    void* mxOs = params.Os_ptr;
    void* mxFs = params.x_ptr;
    void* mxSt = params.St_ptr;
    void* mxdc = params.dc_ptr;
    void* mxdA = params.dA_ptr;
    void const* mxdt = params.delta_ptr;
    void const* mxdb = params.delta_bias_ptr;
    void const* mxA = params.A_ptr;
    void* mxCB = params.CB_ptr;
    void const* mxD = params.D_ptr;
    void const* mxXBC = params.u_ptr;
    void const* mxZ = params.z_ptr;

    if (useTmas[1] || useTmas[3] || useTmas[4])
    {
        // chunk_state
        *(void**) &descs_host[0] = (input_t*) mxXBC + H * P; // B
        *(void**) &descs_host[1] = (input_t*) mxXBC;         // X
        // bmm_chunk
        *(void**) &descs_host[2] = (input_t*) mxXBC + H * P + G * N; // C
        *(void**) &descs_host[3] = (input_t*) mxXBC + H * P;         // B
        // chunk_scan
        *(void**) &descs_host[4] = (input_t*) mxXBC + H * P + G * N; // C
        *(void**) &descs_host[5] = (input_t*) mxOs;
        *(void**) &descs_host[6] = (input_t*) mxCB;
        *(void**) &descs_host[7] = (input_t*) mxXBC; // X

        hipMemcpyAsync(params.desc_ptr, descs_host, sizeof(CUtensorMap) * 8, hipMemcpyHostToDevice, stream);
        hipStreamSynchronize(stream); // to assure hipMemcpyAsync is finished
    }

    CUtensorMap* descs = (CUtensorMap*) params.desc_ptr;

    auto rp = params.remove_padding;
    auto ltip = params.last_token_ids_ptr;
    auto ssmp = params.slot_mapping_ptr;

    hipFuncSetAttribute(reinterpret_cast<const void*>(chunk_cumsum), hipFuncAttributeMaxDynamicSharedMemorySize, shms[0]);
    chunk_cumsum<<<bds[0], tds[0], shms[0], stream>>>(
        B, L, H, P, G, N, mxdc, mxdA, mxdt, mxdb, mxA, mxZ, rp, ltip, dtsp);
    hipFuncSetAttribute(reinterpret_cast<const void*>(chunk_state), hipFuncAttributeMaxDynamicSharedMemorySize, shms[1]);
    chunk_state<<<bds[1], tds[1], shms[1], stream>>>(
        B, L, H, P, G, N, mxSt, mxdc, mxdA, (useTmas[1] ? &descs[0] : mxXBC), rp, ltip);
    hipFuncSetAttribute(reinterpret_cast<const void*>(state_passing), hipFuncAttributeMaxDynamicSharedMemorySize, shms[2]);
    state_passing<<<bds[2], tds[2], shms[2], stream>>>(B, L, H, P, G, N, mxOs, mxFs, mxSt, mxdA, rp, ltip, ssmp);
    hipFuncSetAttribute(reinterpret_cast<const void*>(bmm_chunk), hipFuncAttributeMaxDynamicSharedMemorySize, shms[3]);
    bmm_chunk<<<bds[3], tds[3], shms[3], stream>>>(B, L, H, P, G, N, mxCB, (useTmas[3] ? &descs[2] : mxXBC), rp, ltip);
    hipFuncSetAttribute(reinterpret_cast<const void*>(chunk_scan), hipFuncAttributeMaxDynamicSharedMemorySize, shms[4]);
    chunk_scan<<<bds[4], tds[4], shms[4], stream>>>(
        B, L, H, P, G, N, mxY, mxOs, mxdc, mxdA, mxCB, mxD, (useTmas[4] ? &descs[4] : mxXBC), mxZ, rp, ltip);
}

#define INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(input_t, weight_t)                                                        \
    template void invokeSelectiveScan<input_t, weight_t>(SSMParamsBase & params, hipStream_t stream);

INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(float, float);
INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_SELECTIVE_SCAN_DATA_TYPE

#define INSTANTIATE_CHUNK_SCAN_DATA_TYPE(input_t, weight_t)                                                            \
    template void invokeChunkScan<input_t, weight_t>(                                                                  \
        SSMParamsBase & params, hipStream_t stream, tensorrt_llm::common::CUDADriverWrapper * driver);

INSTANTIATE_CHUNK_SCAN_DATA_TYPE(float, float);
INSTANTIATE_CHUNK_SCAN_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
INSTANTIATE_CHUNK_SCAN_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_CHUNK_SCAN_DATA_TYPE

////////////////////////////////////////////////////////////////////////////////////////////////////

template <typename input_t, typename weight_t, int DSTATE = 16, int CHANNELS_PER_BLOCK = 128, bool MAMBA_V1 = true,
    int STATE_UNROLL = 16>
__launch_bounds__(128, 2) __global__ void selective_scan_update_kernel(SSMParamsBase params)
{

    input_t* output = reinterpret_cast<input_t*>(params.out_ptr);
    input_t* state = reinterpret_cast<input_t*>(params.x_ptr);
    input_t* x = reinterpret_cast<input_t*>(params.u_ptr);
    input_t* dt = reinterpret_cast<input_t*>(params.delta_ptr);
    weight_t* A = reinterpret_cast<weight_t*>(params.A_ptr);
    input_t* B = reinterpret_cast<input_t*>(params.BC_ptr);
    input_t* C = reinterpret_cast<input_t*>(params.BC_ptr);
    weight_t* D = reinterpret_cast<weight_t*>(params.D_ptr);
    input_t* z = reinterpret_cast<input_t*>(params.z_ptr);
    weight_t* dt_bias = reinterpret_cast<weight_t*>(params.delta_bias_ptr);
    bool dt_softplus = params.delta_softplus;
    int num_channels = params.dim;
    int nheads = params.nheads;
    int ngroups = params.ngroups;

    int const channel = blockIdx.x * blockDim.x + threadIdx.x;
    if (channel >= num_channels)
        return;
    int const sample = blockIdx.y;
    int const head_dim = num_channels / nheads;
    int const head = channel / head_dim;
    int const head_chl = channel % head_dim;
    int const group = head / (nheads / ngroups);

    int const slot_idx = params.slot_mapping_ptr == nullptr ? sample : params.slot_mapping_ptr[sample];
    int const dt_d_idx = MAMBA_V1 ? channel : head;
    int const bc_dim = MAMBA_V1 ? 2 * DSTATE : 2 * ngroups * params.dstate;
    int const x_dim = MAMBA_V1 ? num_channels : num_channels + bc_dim;
    int const z_dim = MAMBA_V1 ? num_channels : 2 * num_channels + bc_dim + (nheads + 7) / 8 * 8;
    int const dt_dim = MAMBA_V1 ? num_channels : (z ? z_dim : z_dim - num_channels);
    int const dt_offset = MAMBA_V1 ? sample * dt_dim : sample * dt_dim + dt_dim - (nheads + 7) / 8 * 8;
    int const bc_offset = MAMBA_V1 ? sample * (bc_dim + params.dt_rank) : sample * (num_channels + bc_dim);
    int const b_offset = MAMBA_V1 ? params.dt_rank : num_channels + params.dstate * group;
    int const c_offset = MAMBA_V1 ? params.dt_rank + DSTATE : num_channels + params.dstate * (ngroups + group);

    input_t* my_state = &state[slot_idx * num_channels * (MAMBA_V1 ? DSTATE : params.dstate)];
    input_t* my_output = &output[sample * num_channels];

    int const state_loops = ((MAMBA_V1 ? DSTATE : params.dstate) + STATE_UNROLL - 1) / STATE_UNROLL;

    float my_x, my_dt, my_z, my_dt_bias, out;
    my_x = toFloat(x[sample * x_dim + channel]);
    my_z = z ? toFloat(z[sample * z_dim + channel]) : 0.f;
    my_dt = toFloat(dt[dt_offset + dt_d_idx]);
    my_dt_bias = dt_bias ? toFloat(dt_bias[dt_d_idx]) : 0.f;
    out = D ? toFloat(D[dt_d_idx]) * my_x : 0.f;

    float dt_b = my_dt + my_dt_bias;
    float dt_b_sp = 1.0f;
    if (dt_softplus)
    {
        dt_b_sp = dt_b <= 20.f ? __logf(1.f + __expf(dt_b)) : dt_b; // softplus
    }

    if (MAMBA_V1)
    {
        float rA[DSTATE];
        float rB[DSTATE];
        float rC[DSTATE];
        float rState[DSTATE];
#pragma unroll
        for (int i = 0; i < DSTATE; i++)
        {
            rA[i] = toFloat(A[i * num_channels + channel]);
            rB[i] = toFloat(B[bc_offset + b_offset + i]);
            rC[i] = toFloat(C[bc_offset + c_offset + i]);
            rState[i] = toFloat(my_state[i * num_channels + channel]);
        }
#pragma unroll
        for (int i = 0; i < DSTATE; i++)
        {
            float dA = __expf(rA[i] * dt_b_sp);
            float dB = rB[i] * dt_b_sp;
            float sdA = rState[i] * dA;
            float dBx = dB * my_x;
            float newState = sdA + dBx;
            // Write the new state back out to the cache
            convertAndStore(&my_state[i * num_channels + channel], newState);
            out += newState * rC[i];
        }
    }
    else
    {
        float A_tmp = toFloat(A[head]);
        float rB[STATE_UNROLL];
        float rC[STATE_UNROLL];
        float rState[STATE_UNROLL];
        for (int si = 0; si < state_loops; si++)
        {
            int i_offset = si * STATE_UNROLL;
#pragma unroll
            for (int i = 0; i < STATE_UNROLL; i++)
            {
                rB[i] = toFloat(B[bc_offset + b_offset + i_offset + i]);
                rC[i] = toFloat(C[bc_offset + c_offset + i_offset + i]);
                rState[i] = toFloat(
                    my_state[(head * (MAMBA_V1 ? DSTATE : params.dstate) + i_offset + i) * head_dim + head_chl]);
            }
#pragma unroll
            for (int i = 0; i < STATE_UNROLL; i++)
            {
                float dA = __expf(A_tmp * dt_b_sp);
                float dB = rB[i] * dt_b_sp;
                float sdA = rState[i] * dA;
                float dBx = dB * my_x;
                float newState = sdA + dBx;
                // Write the new state back out to the cache
                convertAndStore(
                    &my_state[(head * (MAMBA_V1 ? DSTATE : params.dstate) + i_offset + i) * head_dim + head_chl],
                    newState);
                out += newState * rC[i];
            }
        }
    }

    if (z)
    {
        float sig_z = __fdividef(1.f, (1.f + __expf(0.f - my_z)));
        float silu_z = my_z * sig_z;
        out *= silu_z;
    }

    convertAndStore(&my_output[channel], out);
}

template <typename input_t, typename weight_t>
void invokeSelectiveScanUpdate(SSMParamsBase& params, hipStream_t stream)
{
    int samples = params.batch;
    int channels = params.dim;
    int nheads = params.nheads;
    int ngroups = params.ngroups;

    int const threads = 128;
    int const blocks = (channels + threads - 1) / threads;
    dim3 block(threads, 1);
    dim3 grid(blocks, samples);

    TLLM_CHECK_WITH_INFO(nheads % ngroups == 0, "nheads must be divisible by ngroups");
    if (params.is_mamba2)
    {
        TLLM_CHECK(params.dstate % 16 == 0);
        selective_scan_update_kernel<input_t, weight_t, 128, 128, false><<<grid, block, 0, stream>>>(params);
    }
    else
    {
        TLLM_CHECK(params.dstate == 16);
        selective_scan_update_kernel<input_t, weight_t, 16, 128, true><<<grid, block, 0, stream>>>(params);
    }
}

#define INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(input_t, weight_t)                                                 \
    template void invokeSelectiveScanUpdate<input_t, weight_t>(SSMParamsBase & params, hipStream_t stream)

INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(float, float);
INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(half, float);
#ifdef ENABLE_BF16
INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE(__hip_bfloat16, float);
#endif
#undef INSTANTIATE_SELECTIVE_SCAN_UPDATE_DATA_TYPE

} // namespace kernels
} // namespace tensorrt_llm
