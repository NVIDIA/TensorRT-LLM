#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/common/quantTypeUtils.cuh"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/quantization.cuh"
#include "tensorrt_llm/kernels/quantization.h"
#include <float.h>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T>
void invokeQuantization(
    int8_t* dst, T const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize)
{
    TLLM_CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    dim3 grid(std::min(numBlocks, maxGridSize));
    TLLM_CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (float4 const*) src, size / 4, scalePtr);
    }
    else if (std::is_same_v<T, half>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (half2 const*) src, size / 4, scalePtr);
    }
#ifdef ENABLE_BF16
    else if (std::is_same_v<T, __hip_bfloat16>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (__hip_bfloat162 const*) src, size / 4, scalePtr);
    }
#endif
}

template void invokeQuantization<float>(
    int8_t* dst, float const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize);

template void invokeQuantization<half>(
    int8_t* dst, half const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize);

#ifdef ENABLE_BF16
template void invokeQuantization<__hip_bfloat16>(int8_t* dst, __hip_bfloat16 const* src, int64_t const size,
    float const* scalePtr, hipStream_t stream, int maxGridSize);
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

// Do per-token (row) quantization from fp16/bf16/fp32 to int8/fp8_e4m3.
template <typename T, typename QuantT>
void invokePerTokenQuantization(QuantT* dst, T const* src, int64_t const numRows, int64_t const numCols,
    float const* clampPtr, float* scalePtr, float* sumPtr, QuantMode quantMode, hipStream_t stream)
{
    // each block is responsible for a single row
    dim3 const block(512);
    dim3 const grid(numRows);

    // The number of elements in the packed uint4 vec.
    static constexpr int NUM_ELTS_PER_VEC = sizeof(uint4) / sizeof(T);
    TLLM_CHECK_WITH_INFO(numCols % NUM_ELTS_PER_VEC == 0, "Not supported.");

    // Cache vectors to smem to avoid reloading.
    size_t const dynamicSmemSz = numCols * sizeof(T);
    // Need to check if smem capacity is enough.
    bool useSmem = true;
    if (dynamicSmemSz >= 48 * 1024)
    {
        hipError_t res = hipFuncSetAttribute(reinterpret_cast<const void*>(
            perTokenQuantization<T), QuantT, true>, hipFuncAttributeMaxDynamicSharedMemorySize, dynamicSmemSz);
        // Fall back to reloading-reversion if smem is not enough.
        useSmem = (res == hipSuccess);
    }

    // Enable min_scaling_factor if it is fp8 rowwise per-token quantization.
    bool hasFp8MinScaling = quantMode.hasFp8RowWise();
    // Do we use smem ?
    if (useSmem)
    {
        perTokenQuantization<T, QuantT, true><<<grid, block, dynamicSmemSz, stream>>>(
            dst, src, numRows, numCols, clampPtr, scalePtr, sumPtr, hasFp8MinScaling);
    }
    else
    {
        perTokenQuantization<T, QuantT, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, clampPtr, scalePtr, sumPtr, hasFp8MinScaling);
    }
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T, QuantT)                                                           \
    template void invokePerTokenQuantization(QuantT* dst, const T* src, const int64_t numRows, const int64_t numCols,  \
        float const* clampPtr, float* scalePtr, float* sumPtr, QuantMode quantMode, hipStream_t stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float, int8_t);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half, int8_t);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////
// FP4 Quantization

template <typename T>
void invokeFP4Quantization(int m, int n, T const* input, float const* SFScale, int64_t* output, int32_t* SFOuput,
    bool useUE8M0, int multiProcessorCount, hipStream_t stream)
{
    // Grid, Block size.
    // Each thread converts 8 values.
    dim3 block(std::min(int(n / CVT_FP4_ELTS_PER_THREAD), 512));
    // Get number of blocks per SM (assume we can fully utilize the SM).
    int const numBlocksPerSM = 2048 / block.x;
    dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

    // Launch the cvt kernel.
    if (useUE8M0)
    {
        auto* kernel_instance = &cvt_fp16_to_fp4<T, true>;
        hipLaunchConfig_t config;
        config.gridDim = grid;
        config.blockDim = block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attrs[1];
        attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.numAttrs = 1;
        config.attrs = attrs;
        cudaLaunchKernelEx(&config, kernel_instance, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
            reinterpret_cast<uint32_t*>(SFOuput));
    }
    else
    {
        auto* kernel_instance = &cvt_fp16_to_fp4<T, false>;
        hipLaunchConfig_t config;
        config.gridDim = grid;
        config.blockDim = block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attrs[1];
        attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.numAttrs = 1;
        config.attrs = attrs;
        cudaLaunchKernelEx(&config, kernel_instance, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
            reinterpret_cast<uint32_t*>(SFOuput));
    }
}

#ifdef ENABLE_FP8
template <>
void invokeFP4Quantization(int m, int n, __hip_fp8_e4m3_fnuz const* input, float const* SFScale, int64_t* output,
    int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream)
{
    // Grid, Block size.
    // Each thread converts 16 values.
    dim3 block(std::min(int(n / CVT_FP8_TO_FP4_ELTS_PER_THREAD), 512));
    // Get number of blocks per SM (assume we can fully utilize the SM).
    int const numBlocksPerSM = 2048 / block.x;
    dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

    // Launch the cvt kernel.
    if (useUE8M0)
    {
        cvt_fp8_to_fp4<true><<<grid, block, 0, stream>>>(
            m, n, input, SFScale, reinterpret_cast<uint64_t*>(output), reinterpret_cast<uint32_t*>(SFOuput));
    }
    else
    {
        cvt_fp8_to_fp4<false><<<grid, block, 0, stream>>>(
            m, n, input, SFScale, reinterpret_cast<uint64_t*>(output), reinterpret_cast<uint32_t*>(SFOuput));
    }
}
#endif

template <typename T>
void invokeBatchedFP4Quantization(int b, int m, int n, T const* input, float const* SFScale, int64_t* output,
    int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream)
{
    // Grid, Block size.
    // Each thread converts 8 values.
    dim3 block(std::min(int(n / CVT_FP4_ELTS_PER_THREAD), 512));
    // Get number of blocks per SM (assume we can fully utilize the SM).
    int const numBlocksPerSM = 2048 / block.x;
    dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

    // Launch the cvt kernel.
    if (useUE8M0)
    {
        auto* kernel_instance = &cvt_fp16_to_fp4_3d<T, true>;
        hipLaunchConfig_t config;
        config.gridDim = grid;
        config.blockDim = block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attrs[1];
        attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.numAttrs = 1;
        config.attrs = attrs;
        cudaLaunchKernelEx(&config, kernel_instance, b, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
            reinterpret_cast<uint32_t*>(SFOuput));
    }
    else
    {
        auto* kernel_instance = &cvt_fp16_to_fp4_3d<T, false>;
        hipLaunchConfig_t config;
        config.gridDim = grid;
        config.blockDim = block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attrs[1];
        attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.numAttrs = 1;
        config.attrs = attrs;
        cudaLaunchKernelEx(&config, kernel_instance, b, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
            reinterpret_cast<uint32_t*>(SFOuput));
    }
}

#ifdef ENABLE_FP8
template <>
void invokeBatchedFP4Quantization(int b, int m, int n, __hip_fp8_e4m3_fnuz const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream)
{
    // Grid, Block size.
    // Each thread converts 16 values.
    dim3 block(std::min(int(n / CVT_FP8_TO_FP4_ELTS_PER_THREAD), 512));
    // Get number of blocks per SM (assume we can fully utilize the SM).
    int const numBlocksPerSM = 2048 / block.x;
    dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

    // Launch the cvt kernel.
    if (useUE8M0)
    {
        cvt_fp8_to_fp4_3d<true><<<grid, block, 0, stream>>>(
            b, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output), reinterpret_cast<uint32_t*>(SFOuput));
    }
    else
    {
        cvt_fp8_to_fp4_3d<false><<<grid, block, 0, stream>>>(
            b, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output), reinterpret_cast<uint32_t*>(SFOuput));
    }
}
#endif

// Instantiate the function.
template void invokeFP4Quantization(int m, int n, half const* input, float const* SFScale, int64_t* output,
    int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream);
template void invokeBatchedFP4Quantization(int b, int m, int n, half const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeFP4Quantization(int m, int n, __hip_bfloat16 const* input, float const* SFScale, int64_t* output,
    int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream);
template void invokeBatchedFP4Quantization(int b, int m, int n, __hip_bfloat16 const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream);
#endif
} // namespace kernels
} // namespace tensorrt_llm
