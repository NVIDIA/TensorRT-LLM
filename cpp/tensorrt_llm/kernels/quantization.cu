#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/common/quantTypeUtils.cuh"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/quantization.cuh"
#include "tensorrt_llm/kernels/quantization.h"
#include <float.h>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

template <typename T>
void invokeQuantization(
    int8_t* dst, T const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize)
{
    TLLM_CHECK_WITH_INFO(size % 4 == 0, "[ERROR][invokeQuantization] size should be a multiple of 4.\n");

    int numBlocks{static_cast<int>((size + 255) / 256)};
    dim3 grid(std::min(numBlocks, maxGridSize));
    TLLM_CHECK_WITH_INFO(grid.x <= maxGridSize, "[ERROR][invokeQuantization] grid max size is exceeded\n");
    dim3 block(64);
    if (std::is_same_v<T, float>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (float4 const*) src, size / 4, scalePtr);
    }
    else if (std::is_same_v<T, half>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (half2 const*) src, size / 4, scalePtr);
    }
#ifdef ENABLE_BF16
    else if (std::is_same_v<T, __hip_bfloat16>)
    {
        quantizedKernel<<<grid, block, 0, stream>>>((char4*) dst, (__hip_bfloat162 const*) src, size / 4, scalePtr);
    }
#endif
}

template void invokeQuantization<float>(
    int8_t* dst, float const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize);

template void invokeQuantization<half>(
    int8_t* dst, half const* src, int64_t const size, float const* scalePtr, hipStream_t stream, int maxGridSize);

#ifdef ENABLE_BF16
template void invokeQuantization<__hip_bfloat16>(int8_t* dst, __hip_bfloat16 const* src, int64_t const size,
    float const* scalePtr, hipStream_t stream, int maxGridSize);
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////

// Do per-token (row) quantization from fp16/bf16/fp32 to int8/fp8_e4m3.
template <typename T, typename QuantT>
void invokePerTokenQuantization(QuantT* dst, T const* src, int64_t const numRows, int64_t const numCols,
    float const* clampPtr, float* scalePtr, float* sumPtr, QuantMode quantMode, hipStream_t stream)
{
    // each block is responsible for a single row
    dim3 const block(512);
    dim3 const grid(numRows);

    // The number of elements in the packed uint4 vec.
    static constexpr int NUM_ELTS_PER_VEC = sizeof(uint4) / sizeof(T);
    TLLM_CHECK_WITH_INFO(numCols % NUM_ELTS_PER_VEC == 0, "Not supported.");

    // Cache vectors to smem to avoid reloading.
    size_t const dynamicSmemSz = numCols * sizeof(T);
    // Need to check if smem capacity is enough.
    bool useSmem = true;
    if (dynamicSmemSz >= 48 * 1024)
    {
        hipError_t res = hipFuncSetAttribute(reinterpret_cast<const void*>(
            perTokenQuantization<T), QuantT, true>, hipFuncAttributeMaxDynamicSharedMemorySize, dynamicSmemSz);
        // Fall back to reloading-reversion if smem is not enough.
        useSmem = (res == hipSuccess);
    }

    // Enable min_scaling_factor if it is fp8 rowwise per-token quantization.
    bool hasFp8MinScaling = quantMode.hasFp8RowWise();
    // Do we use smem ?
    if (useSmem)
    {
        perTokenQuantization<T, QuantT, true><<<grid, block, dynamicSmemSz, stream>>>(
            dst, src, numRows, numCols, clampPtr, scalePtr, sumPtr, hasFp8MinScaling);
    }
    else
    {
        perTokenQuantization<T, QuantT, false>
            <<<grid, block, 0, stream>>>(dst, src, numRows, numCols, clampPtr, scalePtr, sumPtr, hasFp8MinScaling);
    }
}

#define INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(T, QuantT)                                                           \
    template void invokePerTokenQuantization(QuantT* dst, const T* src, const int64_t numRows, const int64_t numCols,  \
        float const* clampPtr, float* scalePtr, float* sumPtr, QuantMode quantMode, hipStream_t stream)

INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float, int8_t);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half, int8_t);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16, int8_t);
#endif

#ifdef ENABLE_FP8
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(float, __hip_fp8_e4m3_fnuz);
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(half, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
INSTANTIATE_INVOKE_PER_TOKEN_QUANTIZATION(__hip_bfloat16, __hip_fp8_e4m3_fnuz);
#endif
#endif

////////////////////////////////////////////////////////////////////////////////////////////////////
// FP4 Quantization

template <typename T, int SF_VEC_SIZE>
void invokeFP4Quantization(int m, int n, T const* input, float const* SFScale, int64_t* output, int32_t* SFOuput,
    bool useUE8M0, FP4QuantizationSFLayout layout, int multiProcessorCount, hipStream_t stream)
{
#ifdef ENABLE_FP8
    if constexpr (std::is_same_v<T, __hip_fp8_e4m3_fnuz>)
    {
        // Grid, Block size.
        // Each thread converts 16 values.
        dim3 block(std::min(int(n / CVT_FP8_TO_FP4_ELTS_PER_THREAD), 512));
        // Get number of blocks per SM (assume we can fully utilize the SM).
        int const numBlocksPerSM = std::max(1u, 2048u / block.x);
        dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

        // Launch the cvt kernel.
        auto* kernel_instance = useUE8M0 ? &cvt_fp8_to_fp4<SF_VEC_SIZE, true> : &cvt_fp8_to_fp4<SF_VEC_SIZE, false>;
        kernel_instance<<<grid, block, 0, stream>>>(
            m, n, input, SFScale, reinterpret_cast<uint64_t*>(output), reinterpret_cast<uint32_t*>(SFOuput), layout);
    }
    else
#endif
    {
        // Grid, Block size.
        // Each thread converts 8 values.
        dim3 block(std::min(int(n / CVT_FP4_ELTS_PER_THREAD), 512));
        // Get number of blocks per SM (assume we can fully utilize the SM).
        int const numBlocksPerSM = std::max(1u, 2048u / block.x);
        dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

        // Launch the cvt kernel.
        auto* kernel_instance
            = useUE8M0 ? &cvt_fp16_to_fp4<T, SF_VEC_SIZE, true> : &cvt_fp16_to_fp4<T, SF_VEC_SIZE, false>;
        hipLaunchConfig_t config;
        config.gridDim = grid;
        config.blockDim = block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attrs[1];
        attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.numAttrs = 1;
        config.attrs = attrs;
        cudaLaunchKernelEx(&config, kernel_instance, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
            reinterpret_cast<uint32_t*>(SFOuput), layout);
    }
}

template <typename T, int SF_VEC_SIZE>
void invokeBatchedFP4Quantization(int b, int m, int n, T const* input, float const* SFScale, int64_t* output,
    int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream)
{
#ifdef ENABLE_FP8
    if constexpr (std::is_same_v<T, __hip_fp8_e4m3_fnuz>)
    {
        // Grid, Block size.
        // Each thread converts 16 values.
        dim3 block(std::min(int(n / CVT_FP8_TO_FP4_ELTS_PER_THREAD), 512));
        // Get number of blocks per SM (assume we can fully utilize the SM).
        int const numBlocksPerSM = std::max(1u, 2048u / block.x);
        dim3 grid(std::min(m, multiProcessorCount * numBlocksPerSM));

        // Launch the cvt kernel.
        auto* kernel_instance
            = useUE8M0 ? &cvt_fp8_to_fp4_3d<SF_VEC_SIZE, true> : &cvt_fp8_to_fp4_3d<SF_VEC_SIZE, false>;
        kernel_instance<<<grid, block, 0, stream>>>(b, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
            reinterpret_cast<uint32_t*>(SFOuput), FP4QuantizationSFLayout::SWIZZLED);
    }
    else
#endif
    {
        // Grid, Block size.
        // Each thread converts 8 values.
        dim3 block(std::min(int(n / CVT_FP4_ELTS_PER_THREAD), 512));
        // Get number of blocks per SM (assume we can fully utilize the SM).
        int const numBlocksPerSM = std::max(1u, 2048u / block.x);
        dim3 grid(std::min(int(m), multiProcessorCount * numBlocksPerSM));

        // Launch the cvt kernel.
        auto* kernel_instance
            = useUE8M0 ? &cvt_fp16_to_fp4_3d<T, SF_VEC_SIZE, true> : &cvt_fp16_to_fp4_3d<T, SF_VEC_SIZE, false>;
        hipLaunchConfig_t config;
        config.gridDim = grid;
        config.blockDim = block;
        config.dynamicSmemBytes = 0;
        config.stream = stream;
        hipLaunchAttribute attrs[1];
        attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
        config.numAttrs = 1;
        config.attrs = attrs;
        cudaLaunchKernelEx(&config, kernel_instance, b, m, n, input, SFScale, reinterpret_cast<uint32_t*>(output),
            reinterpret_cast<uint32_t*>(SFOuput), FP4QuantizationSFLayout::SWIZZLED);
    }
}

__global__ void nvfp4_block_scale_interleave_kernel(
    int numBatches, int numRows, int numCols, uint8_t const* SFIn, uint8_t* SFOutput)
{
    constexpr int SF_VEC_SIZE = 16;
    for (int rowIdx = blockIdx.x; rowIdx < numRows; rowIdx += gridDim.x)
    {
        for (int batchIdx = 0; batchIdx < numBatches; batchIdx++)
        {
            for (int colIdx = threadIdx.x; colIdx < numCols; colIdx += blockDim.x)
            {
                int64_t inOffset = batchIdx * numRows * numCols + rowIdx * numCols + colIdx;
                auto sf = SFIn[inOffset];

                std::optional<int> batchIdxOpt = batchIdx;
                std::optional<int> numRowsOpt = numRows;

                // Without batching, the math in get_sf_out_offset is the same as
                // int const numSfTilesK = (numCols + 4 - 1) / 4;
                // int const tileOffset = ((mi / 128) * numSfTilesK + ki / 4) * 512;
                // int const dstIdx = tileOffset + (mi % 32) * 16 + ((mi % 128) / 32) * 4 + ki % 4;
                auto dstIdx = get_sf_out_offset_128x4<SF_VEC_SIZE>(
                    batchIdxOpt, rowIdx, colIdx, numRowsOpt, numCols * SF_VEC_SIZE);
                SFOutput[dstIdx] = sf;
            }
        }
    }
}

__global__ void nvfp4_block_scale_interleave_reverse_kernel(
    int numBatches, int numRows, int numCols, uint8_t const* SFIn, uint8_t* SFOutput)
{
    constexpr int SF_VEC_SIZE = 16;
    for (int rowIdx = blockIdx.x; rowIdx < numRows; rowIdx += gridDim.x)
    {
        for (int batchIdx = 0; batchIdx < numBatches; batchIdx++)
        {
            for (int colIdx = threadIdx.x; colIdx < numCols; colIdx += blockDim.x)
            {
                std::optional<int> batchIdxOpt = batchIdx;
                std::optional<int> numRowsOpt = numRows;

                // Get the swizzled input index using the same swizzling pattern
                auto srcIdx = get_sf_out_offset_128x4<SF_VEC_SIZE>(
                    batchIdxOpt, rowIdx, colIdx, numRowsOpt, numCols * SF_VEC_SIZE);
                auto sf = SFIn[srcIdx];

                // Output goes to linear layout
                int64_t outOffset = batchIdx * numRows * numCols + rowIdx * numCols + colIdx;
                SFOutput[outOffset] = sf;
            }
        }
    }
}

// This is intended for weight loading, so m and n are large, b <= 256
void invokeNVFP4BlockScaleInterleave(
    int b, int m, int n, uint8_t const* SFIn, uint8_t* SFOutput, int multiProcessorCount, hipStream_t stream)
{
    // Each thread reads 1 int8 value
    dim3 block(std::min(n, 1024));
    // Get number of blocks per SM (assume we can fully utilize the SM).
    int const numBlocksPerSM = std::max(1u, 4096u / block.x);
    dim3 grid(std::min(m, multiProcessorCount * numBlocksPerSM));

    nvfp4_block_scale_interleave_kernel<<<grid, block, 0, stream>>>(b, m, n, SFIn, SFOutput);
}

// This is intended for weight loading, so m and n are large, b <= 256
void invokeNVFP4BlockScaleInterleaveReverse(
    int b, int m, int n, uint8_t const* SFIn, uint8_t* SFOutput, int multiProcessorCount, hipStream_t stream)
{
    // Each thread reads 1 int8 value
    dim3 block(std::min(n, 1024));
    // Get number of blocks per SM (assume we can fully utilize the SM).
    int const numBlocksPerSM = std::max(1u, 4096u / block.x);
    dim3 grid(std::min(m, multiProcessorCount * numBlocksPerSM));

    nvfp4_block_scale_interleave_reverse_kernel<<<grid, block, 0, stream>>>(b, m, n, SFIn, SFOutput);
}

// Instantiate the function.
template void invokeFP4Quantization<half, 16>(int m, int n, half const* input, float const* SFScale, int64_t* output,
    int32_t* SFOuput, bool useUE8M0, FP4QuantizationSFLayout layout, int multiProcessorCount, hipStream_t stream);
template void invokeFP4Quantization<half, 32>(int m, int n, half const* input, float const* SFScale, int64_t* output,
    int32_t* SFOuput, bool useUE8M0, FP4QuantizationSFLayout layout, int multiProcessorCount, hipStream_t stream);
template void invokeBatchedFP4Quantization<half, 16>(int b, int m, int n, half const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream);
template void invokeBatchedFP4Quantization<half, 32>(int b, int m, int n, half const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeFP4Quantization<__hip_bfloat16, 16>(int m, int n, __hip_bfloat16 const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, FP4QuantizationSFLayout layout, int multiProcessorCount,
    hipStream_t stream);
template void invokeFP4Quantization<__hip_bfloat16, 32>(int m, int n, __hip_bfloat16 const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, FP4QuantizationSFLayout layout, int multiProcessorCount,
    hipStream_t stream);
template void invokeBatchedFP4Quantization<__hip_bfloat16, 16>(int b, int m, int n, __hip_bfloat16 const* input,
    float const* SFScale, int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount,
    hipStream_t stream);
template void invokeBatchedFP4Quantization<__hip_bfloat16, 32>(int b, int m, int n, __hip_bfloat16 const* input,
    float const* SFScale, int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount,
    hipStream_t stream);
#endif

#ifdef ENABLE_FP8
template void invokeFP4Quantization<__hip_fp8_e4m3_fnuz, 16>(int m, int n, __hip_fp8_e4m3_fnuz const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, FP4QuantizationSFLayout layout, int multiProcessorCount,
    hipStream_t stream);
template void invokeFP4Quantization<__hip_fp8_e4m3_fnuz, 32>(int m, int n, __hip_fp8_e4m3_fnuz const* input, float const* SFScale,
    int64_t* output, int32_t* SFOuput, bool useUE8M0, FP4QuantizationSFLayout layout, int multiProcessorCount,
    hipStream_t stream);
template void invokeBatchedFP4Quantization<__hip_fp8_e4m3_fnuz, 16>(int b, int m, int n, __hip_fp8_e4m3_fnuz const* input,
    float const* SFScale, int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount,
    hipStream_t stream);
template void invokeBatchedFP4Quantization<__hip_fp8_e4m3_fnuz, 32>(int b, int m, int n, __hip_fp8_e4m3_fnuz const* input,
    float const* SFScale, int64_t* output, int32_t* SFOuput, bool useUE8M0, int multiProcessorCount,
    hipStream_t stream);
#endif

} // namespace kernels
} // namespace tensorrt_llm
