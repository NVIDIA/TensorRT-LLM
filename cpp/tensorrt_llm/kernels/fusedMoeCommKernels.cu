#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/kernels/fusedMoeCommKernels.h"

#include <type_traits>

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/kernels/quantization.cuh"

namespace tensorrt_llm
{
namespace kernels
{

// Quantize a contiguous shared-memory buffer containing elements of DType into NVFP4 with per-16-element FP8 scales.
// Output layout (repeated per 16-element group per lane), followed by one global scale float:
//   [WARP_SIZE * 8 bytes packed e2m1 values] [WARP_SIZE * 1 byte E4M3 per-group scales] ... [global_scale (4 bytes)]
// Each lane writes one 64-bit packed e2m1 for its 16 values and one 1-byte E4M3 scale per group.
// Global scale is computed as (448*6)/absmax and written once at the end of the buffer.
template <typename DType>
__device__ __forceinline__ void quantize_nvfp4_sharedmem(uint8_t* compact_ptr, int sizeInBytes, int laneId)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
    int const numElems = sizeInBytes / sizeof(DType);
    assert(numElems % 2 == 0);
    if (numElems <= 0)
    {
        return;
    }

    DType const* in = reinterpret_cast<DType const*>(compact_ptr);

    // 1) Global absmax across the field (warp reduce) in original dtype precision when possible
    float threadMaxFloat = 0.f;
    if constexpr (std::is_same_v<DType, half> || std::is_same_v<DType, __hip_bfloat16>)
    {
        using DType2 = typename tensorrt_llm::common::packed_as<DType, 2>::type;
        DType2 const* in2 = reinterpret_cast<DType2 const*>(in);
        int const numPairs = numElems / 2;

        // Initialize to zero to avoid a concentrated shared-memory read from index 0 across all lanes
        DType2 localMax2;
        localMax2.x = DType(0.);
        localMax2.y = DType(0.);
        // stride over pairs
        for (int i = laneId; i < numPairs; i += WARP_SIZE)
        {
            DType2 v2 = in2[i];
            localMax2 = tensorrt_llm::common::cuda_max(localMax2, tensorrt_llm::common::cuda_abs(v2));
        }
        // Reduce vector to scalar float in-thread
        DType localMax = tensorrt_llm::common::cuda_max<DType, DType2>(localMax2);
        threadMaxFloat = tensorrt_llm::common::cuda_cast<float>(localMax);
    }
    else
    {
        float localMax = 0.f;
        for (int i = laneId; i < numElems; i += WARP_SIZE)
        {
            float v = fabsf(tensorrt_llm::common::cuda_cast<float>(in[i]));
            localMax = fmaxf(localMax, v);
        }
        threadMaxFloat = localMax;
    }

#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1)
    {
        threadMaxFloat = fmaxf(threadMaxFloat, __shfl_xor_sync(0xffffffff, threadMaxFloat, offset));
    }
    float const eps = 1e-12f;
    float const globalAbsMax = fmaxf(threadMaxFloat, eps);

    // 2) Global scale
    float const SFScaleVal = (448.0f * 6.0f) * (1.0f / globalAbsMax);

    // 3) Output layout
    int const numGroups = (numElems + WARP_SIZE * 16 - 1) / (WARP_SIZE * 16);

    // 8 bytes for e2m1, 1 byte for scale
    int const outputBlockSizeInBytes = 8 * WARP_SIZE + WARP_SIZE;
    uint8_t* const globalScaleOutBytes = compact_ptr + numGroups * outputBlockSizeInBytes;

    // 4) Per-16 group quantization
    int const swizzle_idy = laneId / 4;
    int const swizzle_idx = (laneId % 4) * 8;

    for (int groupId = 0; groupId < numGroups; groupId++)
    {
        int groupStart = groupId * (WARP_SIZE * 16);
        float vecMax = 0.f;
        float2 raw[8];

        if constexpr (std::is_same_v<DType, half> || std::is_same_v<DType, __hip_bfloat16>)
        {
            using DType2 = typename tensorrt_llm::common::packed_as<DType, 2>::type;
            int const numPairs = numElems / 2;
            DType2 const* in2Ptr = reinterpret_cast<DType2 const*>(in);
            int const pairBase = groupStart >> 1;

#pragma unroll
            for (int i = 0; i < 8; ++i)
            {
                int const pi = pairBase + swizzle_idy * 32 + swizzle_idx + (i + swizzle_idy) % 8;
                if (pi < numPairs)
                {
                    DType2 v2 = in2Ptr[pi];
                    float x = tensorrt_llm::common::cuda_cast<float>(v2.x);
                    float y = tensorrt_llm::common::cuda_cast<float>(v2.y);
                    raw[i] = make_float2(x, y);
                    vecMax = fmaxf(vecMax, fmaxf(fabsf(x), fabsf(y)));
                }
                else
                {
                    raw[i] = make_float2(0.0f, 0.0f);
                }
            }
        }
        else
        {
            groupStart += laneId * 16;
#pragma unroll
            for (int i = 0; i < 8; ++i)
            {
                int idx = groupStart + (i << 1);
                if (idx < numElems)
                {
                    float x = tensorrt_llm::common::cuda_cast<float>(in[idx]);
                    float y = (idx + 1 < numElems) ? tensorrt_llm::common::cuda_cast<float>(in[idx + 1]) : 0.0f;
                    raw[i] = make_float2(x, y);
                    vecMax = fmaxf(vecMax, fmaxf(fabsf(x), fabsf(y)));
                }
                else
                {
                    raw[i] = make_float2(0.0f, 0.0f);
                }
            }
        }

        // SF from vecMax and global scale; write as E4M3
        float SFValue = SFScaleVal * (vecMax * reciprocal_approximate_ftz(6.0f));
        __hip_fp8_e4m3_fnuz sf8 = __hip_fp8_e4m3_fnuz(SFValue);
        float SFValueNarrow = static_cast<float>(sf8);
        float const outputScale = (vecMax != 0.f)
            ? reciprocal_approximate_ftz(SFValueNarrow * reciprocal_approximate_ftz(SFScaleVal))
            : 0.0f;

        // Pack 16 values -> 8 bytes e2m1 (use raw[] read above to avoid a second shared-memory read)
        float2 fp2Vals[8];
#pragma unroll
        for (int i = 0; i < 8; ++i)
        {
            fp2Vals[i] = make_float2(raw[i].x * outputScale, raw[i].y * outputScale);
        }
        uint64_t const e2m1Vec = fp32_vec_to_e2m1(fp2Vals);

        uint8_t* const outValPtr = compact_ptr + groupId * outputBlockSizeInBytes + laneId * sizeof(uint64_t);
        uint8_t* const outScalePtr
            = compact_ptr + groupId * outputBlockSizeInBytes + WARP_SIZE * sizeof(uint64_t) + laneId * sizeof(uint8_t);

        if (laneId < 16)
        {
            reinterpret_cast<uint64_t*>(outValPtr)[0] = e2m1Vec;
        }
        __syncwarp();
        if (laneId >= 16)
        {
            reinterpret_cast<uint64_t*>(outValPtr)[0] = e2m1Vec;
        }
        outScalePtr[0] = sf8.__x;
    }

    // Store global scale (fp32) once with a single 32-bit store. Use lane 0 to avoid races.
    if (laneId == 0)
    {
        *reinterpret_cast<float*>(globalScaleOutBytes) = SFScaleVal;
    }
#endif
}

// Convert one lane's packed 16 e2m1 values (in a 64-bit word) into eight float2 values (16 floats).
// Uses 8 cvt.rn.f16x2.e2m1x2 instructions, one per input byte, to produce eight half2 which are cast to float2.
inline __device__ void e2m1_to_fp32_vec(uint64_t e2m1Vec, float2 (&array)[8])
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
    uint32_t out_fp16[8];
    asm volatile(
        "{\n"
        ".reg .b8 b0;\n"
        ".reg .b8 b1;\n"
        ".reg .b8 b2;\n"
        ".reg .b8 b3;\n"
        ".reg .b8 b4;\n"
        ".reg .b8 b5;\n"
        ".reg .b8 b6;\n"
        ".reg .b8 b7;\n"
        ".reg .b32 lo;\n"
        ".reg .b32 hi;\n"
        "mov.b64 {lo, hi}, %8;\n"
        "mov.b32 {b0, b1, b2, b3}, lo;\n"
        "mov.b32 {b4, b5, b6, b7}, hi;\n"
        "cvt.rn.f16x2.e2m1x2   %0, b0;\n"
        "cvt.rn.f16x2.e2m1x2   %1, b1;\n"
        "cvt.rn.f16x2.e2m1x2   %2, b2;\n"
        "cvt.rn.f16x2.e2m1x2   %3, b3;\n"
        "cvt.rn.f16x2.e2m1x2   %4, b4;\n"
        "cvt.rn.f16x2.e2m1x2   %5, b5;\n"
        "cvt.rn.f16x2.e2m1x2   %6, b6;\n"
        "cvt.rn.f16x2.e2m1x2   %7, b7;\n"
        "}"
        : "=r"(out_fp16[0]), "=r"(out_fp16[1]), "=r"(out_fp16[2]), "=r"(out_fp16[3]), "=r"(out_fp16[4]),
        "=r"(out_fp16[5]), "=r"(out_fp16[6]), "=r"(out_fp16[7])
        : "l"(e2m1Vec));

    array[0] = __half22float2(reinterpret_cast<__half2&>(out_fp16[0]));
    array[1] = __half22float2(reinterpret_cast<__half2&>(out_fp16[1]));
    array[2] = __half22float2(reinterpret_cast<__half2&>(out_fp16[2]));
    array[3] = __half22float2(reinterpret_cast<__half2&>(out_fp16[3]));
    array[4] = __half22float2(reinterpret_cast<__half2&>(out_fp16[4]));
    array[5] = __half22float2(reinterpret_cast<__half2&>(out_fp16[5]));
    array[6] = __half22float2(reinterpret_cast<__half2&>(out_fp16[6]));
    array[7] = __half22float2(reinterpret_cast<__half2&>(out_fp16[7]));
#endif
}

template <typename DType>
__device__ __forceinline__ void dequantize_nvfp4_sharedmem(uint8_t* compact_ptr, int sizeInBytes, int laneId)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 1000)
    int const numElems = sizeInBytes / sizeof(DType);
    if (numElems <= 0)
    {
        return;
    }

    int const numGroups = (numElems + WARP_SIZE * 16 - 1) / (WARP_SIZE * 16);

    // New layout matches quantize: per-group blocks of [8*WARP_SIZE bytes values][WARP_SIZE bytes scales],
    // followed by a single 4-byte global scale at the end.
    int const inputBlockSizeInBytes = 8 * WARP_SIZE + WARP_SIZE;
    uint8_t* const globalScaleOutBytes = compact_ptr + numGroups * inputBlockSizeInBytes;
    float const SFScaleVal = reciprocal_approximate_ftz(*reinterpret_cast<float const*>(globalScaleOutBytes));
    __syncwarp();

    DType* out = reinterpret_cast<DType*>(compact_ptr);

    // Process groups in reverse order to avoid overwriting packed input before it is read
    for (int groupId = numGroups - 1; groupId >= 0; --groupId)
    {
        int const groupStart = laneId * 16 + groupId * (WARP_SIZE * 16);
        // Conflict-free read of packed 64-bit e2m1 values from shared memory:
        // serialize half-warps to avoid lane i and i+16 hitting the same bank in the same cycle.
        uint8_t const* const valBase = compact_ptr + groupId * inputBlockSizeInBytes;
        uint64_t packed = 0ull;
        if (laneId < 16)
        {
            packed = reinterpret_cast<uint64_t const*>(valBase)[laneId];
        }
        __syncwarp();
        if (laneId >= 16)
        {
            packed = reinterpret_cast<uint64_t const*>(valBase)[laneId];
        }

        // Read per-lane 1-byte scales to match quantize access pattern
        uint8_t const* const scalesBase = compact_ptr + groupId * inputBlockSizeInBytes + WARP_SIZE * sizeof(uint64_t);
        uint8_t sfByte = scalesBase[laneId];
        __hip_fp8_e4m3_fnuz sf8;
        sf8.__x = sfByte;
        float const SFValueNarrow = static_cast<float>(sf8);
        float const dequantScale = SFScaleVal * SFValueNarrow;
        __syncwarp();

        float2 tmp[8];
        e2m1_to_fp32_vec(packed, tmp);

        // Vectorized stores with swizzle to avoid bank conflicts, matching quantize path
        if constexpr (std::is_same_v<DType, half> || std::is_same_v<DType, __hip_bfloat16>)
        {
            using DType2 = typename tensorrt_llm::common::packed_as<DType, 2>::type;
            DType2* out2 = reinterpret_cast<DType2*>(out);
            int const numPairs = numElems / 2;
            int const pairBase = (groupId * (WARP_SIZE * 16)) >> 1;
            int const swizzle_idy = laneId / 4;
            int const swizzle_idx = (laneId % 4) * 8;

#pragma unroll
            for (int t = 0; t < 8; ++t)
            {
                int const pi = pairBase + swizzle_idy * 32 + swizzle_idx + (t + swizzle_idy) % 8;
                if (pi < numPairs)
                {
                    DType2 v2;
                    v2.x = tensorrt_llm::common::cuda_cast<DType>(tmp[t].x * dequantScale);
                    v2.y = tensorrt_llm::common::cuda_cast<DType>(tmp[t].y * dequantScale);
                    out2[pi] = v2;
                }
            }
        }
        else
        {
            // Fallback linear layout for non-16-bit types
#pragma unroll
            for (int t = 0; t < 8; ++t)
            {
                int idx0 = groupStart + (t << 1);
                if (idx0 < numElems)
                {
                    using DType2 = typename tensorrt_llm::common::packed_as<DType, 2>::type;
                    DType2 v2;
                    v2.x = tensorrt_llm::common::cuda_cast<DType>(tmp[t].x * dequantScale);
                    v2.y = tensorrt_llm::common::cuda_cast<DType>(tmp[t].y * dequantScale);
                    reinterpret_cast<DType2*>(out + idx0)[0] = v2;
                }
            }
        }
        __syncwarp();
    }
#endif
}

static __device__ __forceinline__ uint32_t __as_ptr_smem(void const* __ptr)
{
    // Consider adding debug asserts here.
    return static_cast<uint32_t>(__cvta_generic_to_shared(__ptr));
}

static __device__ __forceinline__ uint64_t __as_ptr_gmem(void const* __ptr)
{
    // Consider adding debug asserts here.
    return static_cast<uint64_t>(__cvta_generic_to_global(__ptr));
}

__device__ __forceinline__ void fence_release_sys()
{
    asm volatile("fence.release.sys;" : : : "memory");
}

__device__ __forceinline__ void mbarrier_init(uint64_t* addr, uint32_t const& count)
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 800
    asm("mbarrier.init.shared.b64 [%0], %1;" : : "r"(__as_ptr_smem(addr)), "r"(count) : "memory");
#endif
}

__device__ __forceinline__ void mbarrier_expect_tx(uint64_t* addr, const uint32_t txCount)
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 900
    asm("mbarrier.expect_tx.relaxed.cta.shared::cta.b64 [%0], %1;"
        :
        : "r"(__as_ptr_smem(addr)), "r"(txCount)
        : "memory");
#endif
}

__device__ __forceinline__ uint64_t mbarrier_arrive(uint64_t* addr)
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 800
    uint64_t state;
    asm("mbarrier.arrive.shared.b64 %0, [%1];" : "=l"(state) : "r"(__as_ptr_smem(addr)) : "memory");
    return state;
#else
    return 0;
#endif
}

__device__ __forceinline__ uint64_t mbarrier_arrive_expect_tx(uint64_t* addr, const uint32_t txCount)
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 900
    uint64_t state;
    asm("mbarrier.arrive.expect_tx.release.cta.shared::cta.b64 %0, [%1], %2;"
        : "=l"(state)
        : "r"(__as_ptr_smem(addr)), "r"(txCount)
        : "memory");
    return state;
#else
    return 0;
#endif
}

__device__ __forceinline__ bool mbarrier_try_wait_parity(uint64_t* addr, uint32_t const& phaseParity)
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 900
    uint32_t waitComplete;
    asm("{\n\t .reg .pred P_OUT; \n\t"
        "mbarrier.try_wait.parity.shared::cta.b64  P_OUT, [%1], %2;\n\t"
        "selp.b32 %0, 1, 0, P_OUT; \n"
        "}"
        : "=r"(waitComplete)
        : "r"(__as_ptr_smem(addr)), "r"(phaseParity)
        : "memory");
    return static_cast<bool>(waitComplete);
#else
    return false;
#endif
}

template <int COPY_SIZE = 4>
__device__ __forceinline__ void ldgsts(int* dstShm, int const* srcMem, bool predGuard)
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 800
    asm volatile(
        "{\n"
        "  .reg .pred p;\n"
        "  setp.ne.b32 p, %0, 0;\n"
        "  @p cp.async.ca.shared.global [%1], [%2], %3;\n"
        "}\n" ::"r"((int) predGuard),
        "r"(__as_ptr_smem(dstShm)), "l"(__as_ptr_gmem(srcMem)), "n"(COPY_SIZE));
#endif
}

__device__ __forceinline__ void cp_async_commit_group()
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 800
    asm volatile("cp.async.commit_group;" : : :);
#endif
}

template <int N = 0>
__device__ __forceinline__ void cp_async_wait_group()
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 800
    asm volatile("cp.async.wait_group %0;" : : "n"(N) : "memory");
#endif
}

__device__ __forceinline__ void cp_async_bulk_g2s(void* dstMem, void const* srcMem, int copySize, uint64_t* smemBar)
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 900
    asm("cp.async.bulk.shared::cta.global.mbarrier::complete_tx::bytes [%0], [%1], %2, [%3];"
        :
        : "r"(__as_ptr_smem(dstMem)), "l"(__as_ptr_gmem(srcMem)), "r"(copySize), "r"(__as_ptr_smem(smemBar))
        : "memory");
#endif
}

__device__ __forceinline__ void cp_async_bulk_s2g(void* dstMem, void const* srcMem, int copySize)
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 900
    asm("cp.async.bulk.global.shared::cta.bulk_group [%0], [%1], %2;"
        :
        : "l"(__as_ptr_gmem(dstMem)), "r"(__as_ptr_smem(srcMem)), "r"(copySize)
        : "memory");
#endif
}

__device__ __forceinline__ void cp_async_bulk_commit_group()
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 900
    asm volatile("cp.async.bulk.commit_group;" : : :);
#endif
}

template <int N = 0>
__device__ __forceinline__ void cp_async_bulk_wait_group()
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 900
    asm volatile("cp.async.bulk.wait_group %0;" : : "n"(N) : "memory");
#endif
}

template <int N = 0>
__device__ __forceinline__ void cp_async_bulk_wait_group_read()
{
#if defined(__HIPCC__) && __CUDA_ARCH__ >= 900
    asm volatile("cp.async.bulk.wait_group.read %0;" : : "n"(N) : "memory");
#endif
}

__host__ void MoeCommFieldInfo::fillFieldInfo(
    uint8_t* dataPtr, size_t elementSize, int vectorSize, int stride, hipDataType dataType)
{
    TLLM_CHECK(elementSize == 1 || elementSize == 2 || elementSize == 4 || elementSize == 8 || elementSize == 16);

    dataPtrBase = dataPtr;

    uint64_t dataPtrU64 = reinterpret_cast<uint64_t>(dataPtr);

    while (elementSize < 16 && dataPtrU64 % (elementSize * 2) == 0 && vectorSize % 2 == 0 && stride % 2 == 0)
    {
        elementSize *= 2;
        vectorSize /= 2;
        stride /= 2;
    }

    if (elementSize == 16)
    {
        alignedUnitBit = 4;
    }
    else if (elementSize == 8)
    {
        alignedUnitBit = 3;
    }
    else if (elementSize == 4)
    {
        alignedUnitBit = 2;
    }
    else if (elementSize == 2)
    {
        alignedUnitBit = 1;
    }
    else
    {
        alignedUnitBit = 0;
    }

    alignedUnitCount = vectorSize;
    alignedUnitStride = stride;
    originalDataType = dataType;
}

class Ll128Proto
{
public:
    static constexpr uint32_t INITIALIZED_VALUE = 0xFFFFFFFFU;

    template <bool USE_FINISH>
    static __device__ __forceinline__ int checkDataReceivedInShm(uint8_t* sharedMemoryBase, uint64_t step,
        int countIn128Bytes, int fifoEntry128ByteIndexBase, int loaded128ByteCount, int warpId, int laneId)
    {
        // return value should be how many package already been received.
        // 0 means no data received, -1 means has received finish package(should be the very first 128 Byte).
        uint64_t* aligned128BytesShm = reinterpret_cast<uint64_t*>(sharedMemoryBase);
        int totalValidCount = 0;
        for (int idxBase = loaded128ByteCount; idxBase < countIn128Bytes; idxBase += WARP_SIZE)
        {
            int idx = idxBase + laneId;
            bool valid = false;
            bool finish = false;
            if (idx < countIn128Bytes)
            {
                int indexInFifoEntry = fifoEntry128ByteIndexBase + idx;
                uint64_t value = aligned128BytesShm[idx * MoeCommFieldInfo::UINT64_PER_128B_BLOCK
                    + indexInFifoEntry % MoeCommFieldInfo::UINT64_PER_128B_BLOCK];
                if (USE_FINISH)
                {
                    finish = (value == (step & (1ULL << 63ULL)));
                    valid = (value == step) || finish;
                }
                else
                {
                    valid = (value == step);
                }
            }
            __syncwarp();
            unsigned validMask = __ballot_sync(WARP_MASK, valid);
            // here we check valid in order, if previous valid is not true, we ignore the current valid.
            int validCount = (validMask == WARP_MASK) ? WARP_SIZE : (__ffs(~validMask) - 1);
            if (USE_FINISH)
            {
                unsigned finishedMask = __ballot_sync(WARP_MASK, finish);
                // finish should be the very first 128 Byte.
                if (finishedMask & 0x1)
                {
                    return -1;
                }
            }
            totalValidCount += validCount;

            if (validCount != WARP_SIZE)
            {
                break;
            }
        }
        return totalValidCount;
    }

    static __device__ __forceinline__ void protoPack(uint8_t* sharedMemoryBase, uint64_t step, int countIn128Bytes,
        int fifoEntry128ByteIndexBase, int warpId, int laneId)
    {
        uint64_t* aligned128BytesShm = reinterpret_cast<uint64_t*>(sharedMemoryBase);
        int halfLaneId = laneId % 16;
        int halfIndex = laneId / 16;
        int tailOffsetIn128Bytes = countIn128Bytes + halfIndex;
        // for LL128 15 * 128 Bytes will be packed to 16 * 128 Bytes, each 16 threads is used for one 15 * 128 bytes.
        for (int idxIn128BytesBase = halfIndex * 15; idxIn128BytesBase < countIn128Bytes; idxIn128BytesBase += 30)
        {
            int tailFlagIndexFromFifoEntry = fifoEntry128ByteIndexBase + tailOffsetIn128Bytes;
            int tailFlagInnerIndex = tailFlagIndexFromFifoEntry % MoeCommFieldInfo::UINT64_PER_128B_BLOCK;
            int idxIn128Bytes = idxIn128BytesBase + halfLaneId;
            int idxFromFifoEntry = fifoEntry128ByteIndexBase + idxIn128Bytes;
            uint64_t tailValue = step;
            uint64_t tailInnerIndex = (halfLaneId >= tailFlagInnerIndex) ? halfLaneId + 1 : halfLaneId;
            if (halfLaneId == 15)
            {
                tailInnerIndex = tailFlagInnerIndex;
            }
            int targetTailIndex = tailOffsetIn128Bytes * MoeCommFieldInfo::UINT64_PER_128B_BLOCK + tailInnerIndex;
            if (idxIn128Bytes < countIn128Bytes && halfLaneId < 15)
            {
                int flagIndex = idxIn128Bytes * MoeCommFieldInfo::UINT64_PER_128B_BLOCK
                    + idxFromFifoEntry % MoeCommFieldInfo::UINT64_PER_128B_BLOCK;
                tailValue = aligned128BytesShm[flagIndex];
                aligned128BytesShm[flagIndex] = step;
            }
            aligned128BytesShm[targetTailIndex] = tailValue;
            tailOffsetIn128Bytes += 2;
        }
        __syncwarp();
    }

    static __device__ __forceinline__ void protoUnpack(uint8_t* sharedMemoryBase, uint64_t step, int countIn128Bytes,
        int fifoEntry128ByteIndexBase, int loaded128ByteCount, int warpId, int laneId)
    {
        uint64_t* aligned128BytesShm = reinterpret_cast<uint64_t*>(sharedMemoryBase);
        int halfLaneId = laneId % 16;
        int halfIndex = laneId / 16;
        int tailOffsetIn128Bytes = countIn128Bytes + halfIndex;
        for (int idxIn128BytesBase = halfIndex * 15; idxIn128BytesBase < countIn128Bytes; idxIn128BytesBase += 30)
        {
            int tailFlagIndexFromFifoEntry = fifoEntry128ByteIndexBase + tailOffsetIn128Bytes;
            int tailFlagInnerIndex = tailFlagIndexFromFifoEntry % MoeCommFieldInfo::UINT64_PER_128B_BLOCK;
            int idxIn128Bytes = idxIn128BytesBase + halfLaneId;
            int idxFromFifoEntry = fifoEntry128ByteIndexBase + idxIn128Bytes;
            uint64_t tailValue = 0;
            int tailInnerIndex = (halfLaneId >= tailFlagInnerIndex) ? halfLaneId + 1 : halfLaneId;
            int targetTailIndex = tailOffsetIn128Bytes * MoeCommFieldInfo::UINT64_PER_128B_BLOCK + tailInnerIndex;
            if (halfLaneId < 15)
            {
                tailValue = aligned128BytesShm[targetTailIndex];
            }
            if (idxIn128Bytes < countIn128Bytes && halfLaneId < 15)
            {
                int flagIndex = idxIn128Bytes * MoeCommFieldInfo::UINT64_PER_128B_BLOCK
                    + idxFromFifoEntry % MoeCommFieldInfo::UINT64_PER_128B_BLOCK;
                aligned128BytesShm[flagIndex] = tailValue;
            }
            tailOffsetIn128Bytes += 2;
        }
        __syncwarp();
    }

    static __device__ __forceinline__ void rearm(
        uint32_t* u32FifoPtr, uint64_t step, int countIn128Bytes, int fifoEntry128ByteIndexBase, int warpId, int laneId)
    {
        // LL128 don't need rearm
    }

    static __device__ __host__ __forceinline__ int computeProtoTransfer128ByteAlignedSize(
        int compact128ByteSizeBeforeProto)
    {
        // each 15 * 128 byte need one tail 128 byte
        int tail128ByteSize = (compact128ByteSizeBeforeProto + 15 * 128 - 1) / (15 * 128) * 128;
        return compact128ByteSizeBeforeProto + tail128ByteSize;
    }
};

using FusedMoeProto = Ll128Proto;

// using FusedMoeProto = LamportProto;

namespace fused_moe_impl
{

// returns copy size for txCount
__device__ __forceinline__ int startFieldG2S(MoeCommFieldInfo const& fieldInfo, int dataIndex,
    uint8_t* sharedMemoryBase, int warpId, int laneId, uint64_t* smemBar)
{
    // we can copy more data than needed, just align to 16 bytes.
    int alignedShmLoadOffset = fieldInfo.getUncompactShmOffset();
    uint8_t* sharedMemoryLoadPtr = sharedMemoryBase + alignedShmLoadOffset;
    int copyByteCount = 0;
    uint8_t* loadPtr = fieldInfo.get16BAlignedLoadCopyRange(dataIndex, &copyByteCount);
    if (laneId == 0 && copyByteCount > 0)
    {
        cp_async_bulk_g2s(sharedMemoryLoadPtr, loadPtr, copyByteCount, smemBar);
    }
    return copyByteCount;
}

__device__ __forceinline__ void startFieldS2G(
    MoeCommFieldInfo const& fieldInfo, int dataIndex, uint8_t* sharedMemoryBase, int warpId, int laneId)
{
    int alignedShmStoreOffset = fieldInfo.getUncompactShmOffset();
    uint8_t* sharedMemoryStorePtr = sharedMemoryBase + alignedShmStoreOffset;
    int copyByteCount = 0;
    int headTailShmIdx;
    int headTailGlobalIdx;
    uint8_t* storePtr
        = fieldInfo.get16BAlignedStoreCopyRange(dataIndex, &copyByteCount, laneId, &headTailShmIdx, &headTailGlobalIdx);
    if (copyByteCount > 0 && laneId == 0)
    {
        cp_async_bulk_s2g(storePtr, sharedMemoryStorePtr + MoeCommFieldInfo::BYTES_PER_16B_BLOCK, copyByteCount);
    }
    if (headTailGlobalIdx >= 0)
    {
        // copy head and tail
        fieldInfo.getRawPtr(dataIndex, nullptr)[headTailGlobalIdx] = sharedMemoryStorePtr[headTailShmIdx];
    }
    __syncwarp();
}

// SRC_AFTER_DST is true, if src > dst, pack will use this,
// SRC_AFTER_DST is false, if src < dst, unpack will use this
template <typename T, bool SRC_AFTER_DST = true>
__device__ __forceinline__ void memmoveSharedMemory(uint8_t* dst, uint8_t const* src, int copySize, int laneId)
{
    int count = (copySize + sizeof(T) - 1) / sizeof(T);
    int warpLoopStart = SRC_AFTER_DST ? 0 : (count + WARP_SIZE - 1) / WARP_SIZE - 1;
    int warpLoopEnd = SRC_AFTER_DST ? (count + WARP_SIZE - 1) / WARP_SIZE : -1;
    int warpLoopUpdate = SRC_AFTER_DST ? 1 : -1;
    for (int i = warpLoopStart; i != warpLoopEnd; i += warpLoopUpdate)
    {
        int idx = laneId + i * WARP_SIZE;
        T data = T{};
        if (idx < count)
        {
            data = reinterpret_cast<T const*>(src)[idx];
        }
        __syncwarp();
        if (idx < count)
        {
            reinterpret_cast<T*>(dst)[idx] = data;
        }
        __syncwarp();
    }
}

template <bool IS_PACK = true>
__device__ __forceinline__ void memmoveFieldOnSharedMemory(
    MoeCommFieldInfo const& fieldInfo, int dataIndex, uint8_t* sharedMemoryBase, int laneId)
{
    int movOffset = fieldInfo.getMemmoveOffsets(dataIndex);
    if (movOffset == 0)
    {
        // if movOffset is 0, src and dst are the same, don't need memmove.
        return;
    }
    int alignedBytes = 1 << fieldInfo.alignedUnitBit;
    int copySize = fieldInfo.alignedUnitCount * alignedBytes;
    uint8_t* sharedMemoryCompact = sharedMemoryBase + fieldInfo.getCompactShmOffset();
    uint8_t* sharedMemoryUncompact = sharedMemoryCompact + movOffset;
    uint8_t* sharedMemoryDst = IS_PACK ? sharedMemoryCompact : sharedMemoryUncompact;
    uint8_t* sharedMemorySrc = IS_PACK ? sharedMemoryUncompact : sharedMemoryCompact;

    if (movOffset % 16 == 0)
    {
        memmoveSharedMemory<int4, IS_PACK>(sharedMemoryDst, sharedMemorySrc, copySize, laneId);
    }
    else if (movOffset % 8 == 0)
    {
        memmoveSharedMemory<int64_t, IS_PACK>(sharedMemoryDst, sharedMemorySrc, copySize, laneId);
    }
    else if (movOffset % 4 == 0)
    {
        memmoveSharedMemory<int, IS_PACK>(sharedMemoryDst, sharedMemorySrc, copySize, laneId);
    }
    else if (movOffset % 2 == 0)
    {
        memmoveSharedMemory<int16_t, IS_PACK>(sharedMemoryDst, sharedMemorySrc, copySize, laneId);
    }
    else
    {
        memmoveSharedMemory<int8_t, IS_PACK>(sharedMemoryDst, sharedMemorySrc, copySize, laneId);
    }
}

template <int FIELD_COUNT = MOE_COMM_FIELD_MAX_COUNT>
__device__ __forceinline__ void packAllFields(
    FusedMoeFieldInfo const& sendFieldInfo, int dataIndex, uint8_t* sharedMemoryBase, int laneId)
{
#pragma unroll
    for (int i = 0; i < FIELD_COUNT; i++)
    {
        memmoveFieldOnSharedMemory<true>(sendFieldInfo.fieldsInfo[i], dataIndex, sharedMemoryBase, laneId);
    }
    __syncwarp();
}

template <int FIELD_COUNT = MOE_COMM_FIELD_MAX_COUNT>
__device__ __forceinline__ void unpackAllFields(
    FusedMoeFieldInfo const& recvFieldInfo, int dataIndex, uint8_t* sharedMemoryBase, int laneId)
{
#pragma unroll
    for (int i = FIELD_COUNT - 1; i >= 0; i--)
    {
        memmoveFieldOnSharedMemory<false>(recvFieldInfo.fieldsInfo[i], dataIndex, sharedMemoryBase, laneId);
    }
    __syncwarp();
}

__device__ __forceinline__ void initSmemBar(uint64_t* smemBar, int laneId)
{
    if (laneId == 0)
    {
        mbarrier_init(smemBar, WARP_SIZE);
    }
    __syncwarp();
}

__device__ __forceinline__ void smemBarWait(uint64_t* smemBar, uint32_t* phaseParity)
{
    while (!mbarrier_try_wait_parity(smemBar, *phaseParity))
    {
    }
    *phaseParity = 1 - *phaseParity;
}

__device__ __forceinline__ void startWorkspaceS2G(
    uint64_t* fifoEntry, uint8_t* sharedMemoryBase, int send128ByteCount, int fifo128ByteOffset, int warpId, int laneId)
{
    int copyByteCount = send128ByteCount * MoeCommFieldInfo::BYTES_PER_128B_BLOCK;
    if (laneId == 0)
    {
        cp_async_bulk_s2g(fifoEntry + fifo128ByteOffset * MoeCommFieldInfo::BYTES_PER_128B_BLOCK / sizeof(int64_t),
            sharedMemoryBase, copyByteCount);
    }
    __syncwarp();
    cp_async_bulk_commit_group();
}

__device__ __forceinline__ void startWorkspaceS2GReg(
    uint64_t* fifoEntry, uint8_t* sharedMemoryBase, int send128ByteCount, int fifo128ByteOffset, int warpId, int laneId)
{
    int copyInt4Count = send128ByteCount * MoeCommFieldInfo::BYTES_PER_128B_BLOCK / sizeof(int4);
    int4* sharedMemoryInt4 = reinterpret_cast<int4*>(sharedMemoryBase);
    uint64_t* fifoPtr = fifoEntry + fifo128ByteOffset * MoeCommFieldInfo::BYTES_PER_128B_BLOCK / sizeof(int64_t);
    int4* fifoPtrInt4 = reinterpret_cast<int4*>(fifoPtr);
#pragma unroll 4
    for (int i = laneId; i < copyInt4Count; i += WARP_SIZE)
    {
        fifoPtrInt4[i] = sharedMemoryInt4[i];
    }
    __syncwarp();
}

__device__ __forceinline__ uint64_t startWorkspaceG2S(uint8_t* sharedMemoryBase, uint64_t* fifoEntry,
    int allLoad128ByteCount, int fifo128ByteOffset, int loaded128ByteCount, uint64_t* smemBar, int warpId, int laneId)
{
    int copyByteCount = (allLoad128ByteCount - loaded128ByteCount) * MoeCommFieldInfo::BYTES_PER_128B_BLOCK;
    if (laneId == 0)
    {
        cp_async_bulk_g2s(sharedMemoryBase + loaded128ByteCount * MoeCommFieldInfo::BYTES_PER_128B_BLOCK,
            fifoEntry
                + (fifo128ByteOffset + loaded128ByteCount) * MoeCommFieldInfo::BYTES_PER_128B_BLOCK / sizeof(int64_t),
            copyByteCount, smemBar);
    }
    return mbarrier_arrive_expect_tx(smemBar, laneId == 0 ? copyByteCount : 0);
}

__device__ __forceinline__ void g2sBasicFields(FusedMoeFieldInfo const& sendFieldInfo,
    MoeExpertParallelInfo const& expertParallelInfo, int dataIndex, uint8_t* sharedMemoryBase, int laneId)
{
    int topK = expertParallelInfo.topK;
    int* tokenSelectedSlotsPtr = sendFieldInfo.getTokenSelectedSlotsPtr(dataIndex, laneId, topK);
    float* scalePtr = sendFieldInfo.getScalePtr(dataIndex, laneId, topK);
    ldgsts<4>(reinterpret_cast<int*>(sharedMemoryBase) + laneId, tokenSelectedSlotsPtr, laneId < topK);
    ldgsts<4>(reinterpret_cast<int*>(sharedMemoryBase) + laneId + topK, reinterpret_cast<int*>(scalePtr),
        laneId < topK && sendFieldInfo.expertScales != nullptr);
}

// May commit 1 group for basic fields(tokenSelectedSlots and scales) if HAS_BASIC_FIELDS is true
// For other fields, use smemBar.
template <bool HAS_BASIC_FIELDS = true, int FIELD_COUNT = MOE_COMM_FIELD_MAX_COUNT>
__device__ __forceinline__ uint64_t g2sAllFields(FusedMoeFieldInfo const& sendFieldInfo,
    MoeExpertParallelInfo const& expertParallelInfo, int dataIndex, uint8_t* sharedMemoryBase, int warpId, int laneId,
    uint64_t* smemBar)
{
    if (HAS_BASIC_FIELDS)
    {
        g2sBasicFields(sendFieldInfo, expertParallelInfo, dataIndex, sharedMemoryBase, laneId);
        cp_async_commit_group();
    }
    int asyncLoadSize = 0;
#pragma unroll
    for (int i = 0; i < FIELD_COUNT; i++)
    {
        asyncLoadSize
            += startFieldG2S(sendFieldInfo.fieldsInfo[i], dataIndex, sharedMemoryBase, warpId, laneId, smemBar);
    }
    return mbarrier_arrive_expect_tx(smemBar, laneId == 0 ? asyncLoadSize : 0);
}

template <bool HAS_BASIC_FIELDS = true>
__device__ __forceinline__ void waitG2SBasicFields()
{
    if (HAS_BASIC_FIELDS)
    {
        cp_async_wait_group<0>();
        __syncwarp();
    }
}

__device__ __forceinline__ void waitG2SOtherFields(uint64_t* memBar, uint32_t* phaseParity)
{
    tensorrt_llm::kernels::fused_moe_impl::smemBarWait(memBar, phaseParity);
}

template <bool HAS_BASIC_FIELDS = true>
__device__ __forceinline__ void waitG2SAllFields(uint64_t* memBar, uint32_t* phaseParity)
{
    waitG2SBasicFields<HAS_BASIC_FIELDS>();
    waitG2SOtherFields(memBar, phaseParity);
}

__device__ __forceinline__ void waitS2GBulkRead()
{
    cp_async_bulk_wait_group_read<0>();
    __syncwarp();
}

__device__ __forceinline__ void s2gBasicFields(FusedMoeFieldInfo const& recvFieldInfo,
    MoeExpertParallelInfo const& expertParallelInfo, int dataIndex, uint8_t* sharedMemoryBase, int warpId, int laneId)
{
    int topK = expertParallelInfo.topK;
    int* tokenSelectedSlotsPtr = recvFieldInfo.getTokenSelectedSlotsPtr(dataIndex, laneId, topK);
    float* scalePtr = recvFieldInfo.getScalePtr(dataIndex, laneId, topK);
    if (laneId < topK)
    {
        int selectedSlot = reinterpret_cast<int*>(sharedMemoryBase)[laneId];
        *tokenSelectedSlotsPtr = selectedSlot;
        if (recvFieldInfo.expertScales != nullptr)
        {
            float scale = reinterpret_cast<float*>(sharedMemoryBase)[laneId + topK];
            *scalePtr = scale;
        }
    }
}

// Will commit 1 group, for all non-basic fields
template <bool HAS_BASIC_FIELDS = true, int FIELD_COUNT = MOE_COMM_FIELD_MAX_COUNT>
__device__ __forceinline__ void s2gAllFields(FusedMoeFieldInfo const& recvFieldInfo,
    MoeExpertParallelInfo const& expertParallelInfo, int dataIndex, uint8_t* sharedMemoryBase, int warpId, int laneId)
{
    if (HAS_BASIC_FIELDS)
    {
        s2gBasicFields(recvFieldInfo, expertParallelInfo, dataIndex, sharedMemoryBase, warpId, laneId);
        __syncwarp();
    }
#pragma unroll
    for (int i = 0; i < FIELD_COUNT; i++)
    {
        startFieldS2G(recvFieldInfo.fieldsInfo[i], dataIndex, sharedMemoryBase, warpId, laneId);
    }
    cp_async_bulk_commit_group();
}

template <int FIELD_COUNT, bool HAS_BASIC_FIELD = true, bool LOW_PRECISION = false>
class SingleChannelCommunicator
{
public:
    __device__ __forceinline__ SingleChannelCommunicator(FusedMoeFieldInfo const& fieldInfo,
        MoeExpertParallelInfo const& expertParallelInfo, MoeSingleCommMeta const& commMeta,
        FusedMoeWorkspace const& workspace, FusedMoeWorldInfo const& worldInfo, FusedMoePairInfo const& pairInfo,
        uint64_t* smemBar, uint8_t* shmemBase)
        : mFieldInfo(fieldInfo)
        , mExpertParallelInfo(expertParallelInfo)
        , mCommMeta(commMeta)
        , mWorkspace(workspace)
        , mWorldInfo(worldInfo)
        , mPairInfo(pairInfo)
        , mSmemBar(smemBar)
        , mShmemBase(shmemBase)
    {
        if constexpr (LOW_PRECISION)
        {
            static_assert(FIELD_COUNT == 1, "Low precision alltoall only support 1 field");
        }

        mWarpId = threadIdx.x / WARP_SIZE;
        mLaneId = threadIdx.x % WARP_SIZE;

        mFifoBasePtr = mWorkspace.getFifoBasePtr(mWorldInfo, mPairInfo);
        mSenderSideFifoInfo = mWorkspace.getSenderSideFifoInfo(mWorldInfo, mPairInfo);
        mReceiverSideFifoInfo = mWorkspace.getReceiverSideFifoInfo(mWorldInfo, mPairInfo);

        mSingleTransfer128ByteCount = mCommMeta.getTransfer128ByteCount();
        mSingleCompactData128ByteCount = mCommMeta.getCompactData128ByteCount();
        // initialize as need new Entry first
        mFifoEntry128ByteIndexBase = kFifoEntry128ByteCount;
        mFifoEntryIndex = -1;

        tensorrt_llm::kernels::fused_moe_impl::initSmemBar(mSmemBar, mLaneId);
    }

    __device__ __forceinline__ uint64_t* getFifoEntryPtr() const
    {
        return mFifoBasePtr + mFifoEntryIndex * kFifoEntrySizeInU64;
    }

    __device__ __forceinline__ bool needNewEntry() const
    {
        return mFifoEntry128ByteIndexBase + mSingleTransfer128ByteCount > kFifoEntry128ByteCount;
    }

    __device__ __forceinline__ void nextToken()
    {
        mFifoEntry128ByteIndexBase += mSingleTransfer128ByteCount;
    }

    __device__ __forceinline__ void senderInitFifo()
    {
        mHead = mSenderSideFifoInfo->head;
        mTail = mSenderSideFifoInfo->tail;
    }

    __device__ __forceinline__ void receiverInitFifo()
    {
        mHead = mReceiverSideFifoInfo->head;
        mTail = mReceiverSideFifoInfo->tail;
    }

    /*
     * Head     | 0 | 1 | 2 | 3 | 4 | 4 | 4 | 4 | 4 | 5 |
     * Tail     | 0 | 0 | 0 | 0 | 0 | 1 | 2 | 3 | 4 | 4 |
     * Writable | Y | Y | Y | Y | N | Y | Y | Y | Y | Y |
     * Readable | N | Y | Y | Y | Y | Y | Y | Y | N | Y |
     */

    __device__ __forceinline__ void waitEntryWritable()
    {
        while (mTail + kFifoDepth <= mHead)
        {
            mTail = mSenderSideFifoInfo->tail;
        }
    }

    __device__ __forceinline__ void updateWriteEntry()
    {
        __syncwarp();
        mSenderSideFifoInfo->head = mHead;
    }

    __device__ __forceinline__ void waitEntryReadable()
    {
        // always readable as long as flag matches.
    }

    __device__ __forceinline__ void updateReadEntry()
    {
        mReceiverSideFifoInfo->tail = mTail;
        mSenderSideFifoInfo->tail = mTail;
    }

    __device__ __forceinline__ void newSendEntry()
    {
        mFifoEntryIndex = mHead % kFifoDepth;
        mFifoEntry128ByteIndexBase = 0;
        waitEntryWritable();
        __syncwarp();
    }

    __device__ __forceinline__ void newReceiveEntry()
    {
        mFifoEntryIndex = mTail % kFifoDepth;
        mFifoEntry128ByteIndexBase = 0;
        waitEntryReadable();
        __syncwarp();
    }

    __device__ __forceinline__ void doSend(int tokenCount, int* sendIndexMapping)
    {
        senderInitFifo();

        int sendIndex = mPairInfo.channel;
        uint32_t phaseParity = 0;
        for (; sendIndex < tokenCount; sendIndex += mPairInfo.runChannelCount)
        {
            int tokenIndex = sendIndexMapping == nullptr ? sendIndex : sendIndexMapping[sendIndex];
            tensorrt_llm::kernels::fused_moe_impl::g2sAllFields<HAS_BASIC_FIELD, FIELD_COUNT>(
                mFieldInfo, mExpertParallelInfo, tokenIndex, mShmemBase, mWarpId, mLaneId, mSmemBar);
            if (needNewEntry())
            {
                if (mFifoEntryIndex >= 0)
                {
                    // not first entry, update FIFO info from last entry.
                    mHead++;
                    updateWriteEntry();
                }
                newSendEntry();
            }
            tensorrt_llm::kernels::fused_moe_impl::waitG2SAllFields<HAS_BASIC_FIELD>(mSmemBar, &phaseParity);
            tensorrt_llm::kernels::fused_moe_impl::packAllFields<FIELD_COUNT>(
                mFieldInfo, tokenIndex, mShmemBase, mLaneId);

            if constexpr (LOW_PRECISION)
            {
                // quantize here.
                int alignedUnitBit = mFieldInfo.fieldsInfo[0].alignedUnitBit;
                int alignedUnitCount = mFieldInfo.fieldsInfo[0].alignedUnitCount;
                int sizeInBytes = alignedUnitCount * (1 << alignedUnitBit);
                uint8_t* sharedMemoryCompact = mShmemBase + mFieldInfo.fieldsInfo[0].getCompactShmOffset();
                hipDataType originalDataType = mFieldInfo.fieldsInfo[0].originalDataType;

                switch (originalDataType)
                {
                case HIP_R_16BF:
                    quantize_nvfp4_sharedmem<__hip_bfloat16>(sharedMemoryCompact, sizeInBytes, mLaneId);
                    break;
                case HIP_R_16F: quantize_nvfp4_sharedmem<half>(sharedMemoryCompact, sizeInBytes, mLaneId); break;
                default: break;
                }
            }

            FusedMoeProto::protoPack(
                mShmemBase, mHead, mSingleCompactData128ByteCount, mFifoEntry128ByteIndexBase, mWarpId, mLaneId);

            tensorrt_llm::kernels::fused_moe_impl::startWorkspaceS2GReg(getFifoEntryPtr(), mShmemBase,
                mSingleTransfer128ByteCount, mFifoEntry128ByteIndexBase, mWarpId, mLaneId);

            // tensorrt_llm::kernels::fused_moe_impl::waitS2GBulkRead();

            nextToken();
        }
        if (mFifoEntry128ByteIndexBase > 0)
        {
            mHead++;
            updateWriteEntry();
        }
    }

    __device__ __forceinline__ void rearmFifoBuffer()
    {
        constexpr int kUint32CountPer128Byte = 128 / sizeof(uint32_t);
        uint32_t* fifoPtr = reinterpret_cast<uint32_t*>(getFifoEntryPtr());
        fifoPtr += mFifoEntry128ByteIndexBase * kUint32CountPer128Byte;

        FusedMoeProto::rearm(fifoPtr, mTail, mSingleTransfer128ByteCount, mFifoEntry128ByteIndexBase, mWarpId, mLaneId);
        __syncwarp();
    }

    __device__ __forceinline__ void doReceive(int tokenCount, int* recvIndexMapping)
    {
        receiverInitFifo();
        int recvIndex = mPairInfo.channel;
        uint32_t phaseParity = 0;
        bool needRelease = false;
        for (; recvIndex < tokenCount; recvIndex += mPairInfo.runChannelCount)
        {
            int tokenIndex = recvIndexMapping == nullptr ? recvIndex : recvIndexMapping[recvIndex];
            int loaded128ByteCount = 0;
            if (needNewEntry())
            {
                if (mFifoEntryIndex >= 0)
                {
                    // not first entry, update FIFO info from last entry.
                    mTail++;
                    needRelease = true;
                }
                newReceiveEntry();
            }
            while (loaded128ByteCount < mSingleTransfer128ByteCount)
            {
                tensorrt_llm::kernels::fused_moe_impl::startWorkspaceG2S(mShmemBase, getFifoEntryPtr(),
                    mSingleTransfer128ByteCount, mFifoEntry128ByteIndexBase, loaded128ByteCount, mSmemBar, mWarpId,
                    mLaneId);
                if (needRelease)
                {
                    updateReadEntry();
                    needRelease = false;
                }
                tensorrt_llm::kernels::fused_moe_impl::smemBarWait(mSmemBar, &phaseParity);
                loaded128ByteCount += FusedMoeProto::template checkDataReceivedInShm<false>(mShmemBase, mTail,
                    mSingleTransfer128ByteCount, mFifoEntry128ByteIndexBase, loaded128ByteCount, mWarpId, mLaneId);
            }

            FusedMoeProto::protoUnpack(mShmemBase, mTail, mSingleCompactData128ByteCount, mFifoEntry128ByteIndexBase,
                loaded128ByteCount, mWarpId, mLaneId);

            if constexpr (LOW_PRECISION)
            {
                int alignedUnitBit = mFieldInfo.fieldsInfo[0].alignedUnitBit;
                int alignedUnitCount = mFieldInfo.fieldsInfo[0].alignedUnitCount;
                int sizeInBytes = alignedUnitCount * (1 << alignedUnitBit);
                uint8_t* sharedMemoryCompact = mShmemBase + mFieldInfo.fieldsInfo[0].getCompactShmOffset();
                hipDataType originalDataType = mFieldInfo.fieldsInfo[0].originalDataType;

                switch (originalDataType)
                {
                case HIP_R_16BF:
                    dequantize_nvfp4_sharedmem<__hip_bfloat16>(sharedMemoryCompact, sizeInBytes, mLaneId);
                    break;
                case HIP_R_16F: dequantize_nvfp4_sharedmem<half>(sharedMemoryCompact, sizeInBytes, mLaneId); break;
                default: break;
                }
            }

            tensorrt_llm::kernels::fused_moe_impl::unpackAllFields<FIELD_COUNT>(
                mFieldInfo, tokenIndex, mShmemBase, mLaneId);
            tensorrt_llm::kernels::fused_moe_impl::s2gAllFields<HAS_BASIC_FIELD, FIELD_COUNT>(
                mFieldInfo, mExpertParallelInfo, tokenIndex, mShmemBase, mWarpId, mLaneId);
            tensorrt_llm::kernels::fused_moe_impl::waitS2GBulkRead();

            rearmFifoBuffer();
            nextToken();
        }
        if (mFifoEntry128ByteIndexBase > 0)
        {
            mTail++;
            updateReadEntry();
        }
    }

private:
    static constexpr int kFifoEntrySizeInU64 = FusedMoeCommunicator::FIFO_ENTRY_BYTES / sizeof(uint64_t);
    static constexpr int kFifoEntry128ByteCount = FusedMoeCommunicator::FIFO_ENTRY_128_BYTE_COUNT;
    static constexpr int kFifoDepth = FusedMoeCommunicator::FIFO_DEPTH;

    FusedMoeFieldInfo mFieldInfo;
    MoeExpertParallelInfo mExpertParallelInfo;
    MoeSingleCommMeta mCommMeta;
    FusedMoeWorkspace mWorkspace;
    FusedMoeWorldInfo mWorldInfo;
    FusedMoePairInfo mPairInfo;
    uint64_t* mSmemBar;
    uint8_t* mShmemBase;

    int mLaneId;
    int mWarpId;

    uint64_t* mFifoBasePtr;
    SenderSideFifoInfo* mSenderSideFifoInfo;
    ReceiverSideFifoInfo* mReceiverSideFifoInfo;

    int64_t mHead;
    int64_t mTail;

    int mSingleTransfer128ByteCount;
    int mSingleCompactData128ByteCount;
    int mFifoEntry128ByteIndexBase;
    int mFifoEntryIndex;
};

template <int FIELD_COUNT = MOE_COMM_FIELD_MAX_COUNT, bool LOW_PRECISION = false>
__global__ void moeAllToAllKernel(FusedMoeCommKernelParam params, FusedMoeWorkspace workspace, bool hasBasicFields)
{
    __shared__ uint64_t allWarpSmemBar[32];
    extern __shared__ int4 allWarpShm[];

    bool isSender = blockIdx.z == 0;
    int runChannelCount = gridDim.y;
    int group = threadIdx.y;
    SendRecvIndices dataIndices = isSender ? params.sendIndices : params.recvIndices;

    FusedMoePairInfo pairInfo;
    int peerRank = blockIdx.x * blockDim.y + group;
    if (peerRank >= params.worldInfo.epInfo.epSize)
    {
        return;
    }
    int tokenCount;
    int* groupStartPtr = dataIndices.getGroupStart(peerRank, tokenCount);
    if (tokenCount == 0)
    {
        return;
    }

    pairInfo.channel = blockIdx.y;
    pairInfo.runChannelCount = runChannelCount;
    pairInfo.senderRank = isSender ? params.worldInfo.epInfo.epRank : peerRank;
    pairInfo.receiverRank = isSender ? peerRank : params.worldInfo.epInfo.epRank;

    if (isSender)
    {
        int singleShmSize = params.sendCommMeta.getSingleShmSize();
        if (hasBasicFields)
        {
            SingleChannelCommunicator<FIELD_COUNT, true, LOW_PRECISION> comm(params.sendFieldInfo,
                params.expertParallelInfo, params.sendCommMeta, workspace, params.worldInfo, pairInfo,
                allWarpSmemBar + group, reinterpret_cast<uint8_t*>(allWarpShm) + singleShmSize * group);
            comm.doSend(tokenCount, groupStartPtr);
        }
        else
        {
            SingleChannelCommunicator<FIELD_COUNT, false, LOW_PRECISION> comm(params.sendFieldInfo,
                params.expertParallelInfo, params.sendCommMeta, workspace, params.worldInfo, pairInfo,
                allWarpSmemBar + group, reinterpret_cast<uint8_t*>(allWarpShm) + singleShmSize * group);
            comm.doSend(tokenCount, groupStartPtr);
        }
    }
    else
    {
        int singleShmSize = params.recvCommMeta.getSingleShmSize();
        if (hasBasicFields)
        {
            SingleChannelCommunicator<FIELD_COUNT, true, LOW_PRECISION> comm(params.recvFieldInfo,
                params.expertParallelInfo, params.recvCommMeta, workspace, params.worldInfo, pairInfo,
                allWarpSmemBar + group, reinterpret_cast<uint8_t*>(allWarpShm) + singleShmSize * group);
            comm.doReceive(tokenCount, groupStartPtr);
        }
        else
        {
            SingleChannelCommunicator<FIELD_COUNT, false, LOW_PRECISION> comm(params.recvFieldInfo,
                params.expertParallelInfo, params.recvCommMeta, workspace, params.worldInfo, pairInfo,
                allWarpSmemBar + group, reinterpret_cast<uint8_t*>(allWarpShm) + singleShmSize * group);
            comm.doReceive(tokenCount, groupStartPtr);
        }
    }
}

int computeMoeAlltoallMaxDynamicSharedMemorySize()
{
    int devId = -1;
    TLLM_CUDA_CHECK(hipGetDevice(&devId));
    hipFuncAttributes attr{};
    TLLM_CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>((void const*)) moeAllToAllKernel<1>));
    int staticSmem = static_cast<int>(attr.sharedSizeBytes);
    int maxPerBlockShmOptin = 0;
    TLLM_CUDA_CHECK(hipDeviceGetAttribute(&maxPerBlockShmOptin, hipDeviceAttributeSharedMemPerBlockOptin, devId));
    return maxPerBlockShmOptin - staticSmem;
}

} // namespace fused_moe_impl

void FusedMoeFieldInfo::fillMetaInfo(
    MoeSingleCommMeta* singleCommMeta, int topK, bool hasScales, bool hasBasicFields, bool isLowPrecision) const
{
    singleCommMeta->singleUncompactAlignedSize = computeSingleUncompactSize(topK, hasScales, hasBasicFields);

    if (isLowPrecision)
    {
        assert(fieldCount == 1);
        assert(fieldsInfo[0].originalDataType == HIP_R_16F || fieldsInfo[0].originalDataType == HIP_R_16BF);

        auto alignment128 = MoeCommFieldInfo::BYTES_PER_128B_BLOCK;

        auto alignedUnitBit = fieldsInfo[0].alignedUnitBit;
        auto alignedUnitCount = fieldsInfo[0].alignedUnitCount;
        auto originalFieldSize = alignedUnitCount * (1 << alignedUnitBit);

        int numElements = originalFieldSize / 2;
        int numGroups = (numElements + WARP_SIZE * 16 - 1) / (WARP_SIZE * 16);
        int sizePerGroupInBytes = (WARP_SIZE * 16 / 2 + WARP_SIZE * 1);

        int totalSize = numGroups * sizePerGroupInBytes + 4;
        int compactSize = (totalSize + alignment128 - 1) / alignment128 * alignment128;

        singleCommMeta->singleCompactAlignedSize = compactSize;
        singleCommMeta->singleTransferAlignedSize
            = FusedMoeProto::computeProtoTransfer128ByteAlignedSize(singleCommMeta->singleCompactAlignedSize);
        return;
    }

    singleCommMeta->singleCompactAlignedSize = computeSingleCompactSize(topK, hasScales, hasBasicFields);
    singleCommMeta->singleTransferAlignedSize
        = FusedMoeProto::computeProtoTransfer128ByteAlignedSize(singleCommMeta->singleCompactAlignedSize);
}

void FusedMoeFieldInfo::fillFieldPlacementInfo(int topK, bool hasBasicFields)
{
    int basicFieldSize = 0;
    if (hasBasicFields)
    {
        basicFieldSize = topK * sizeof(int) + (expertScales != nullptr ? topK * sizeof(float) : 0);
        // align to 16 bytes
        basicFieldSize = (basicFieldSize + MoeCommFieldInfo::BYTES_PER_16B_BLOCK - 1)
            / MoeCommFieldInfo::BYTES_PER_16B_BLOCK * MoeCommFieldInfo::BYTES_PER_16B_BLOCK;
    }
    int offset = basicFieldSize;
    int unalignedFieldIndex = 0;
    for (int i = 0; i < fieldCount; i++)
    {
        fieldsInfo[i].compact16BOffset = offset / MoeCommFieldInfo::BYTES_PER_16B_BLOCK;
        offset += fieldsInfo[i].getFieldCompactSize();
        fieldsInfo[i].unalignedFieldIndex = unalignedFieldIndex;
        if (fieldsInfo[i].alignedUnitBit < 4)
        {
            unalignedFieldIndex++;
        }
    }
    for (int i = fieldCount; i < MOE_COMM_FIELD_MAX_COUNT; i++)
    {
        fieldsInfo[i].setUnused();
    }
}

void FusedMoeWorkspace::initializeLocalWorkspace(FusedMoeWorldInfo const& worldInfo)
{
    int epSize = worldInfo.epInfo.epSize;
    int epRank = worldInfo.epInfo.epRank;
    size_t fifoSize = static_cast<size_t>(FusedMoeCommunicator::FIFO_TOTAL_BYTES) * epSize * channelCount;
    size_t senderSideInfoSize = sizeof(SenderSideFifoInfo) * epSize * channelCount;
    size_t receiverSideInfoSize = sizeof(ReceiverSideFifoInfo) * epSize * channelCount;
    uint64_t* localWorkspacePtr = workspacePtr + epRank * rankStrideInU64;
    TLLM_CU_CHECK(hipMemsetD32(reinterpret_cast<hipDeviceptr_t>(localWorkspacePtr), FusedMoeProto::INITIALIZED_VALUE,
        fifoSize / sizeof(uint32_t)));
    TLLM_CUDA_CHECK(hipMemset(
        reinterpret_cast<uint8_t*>(localWorkspacePtr) + fifoSize, 0, senderSideInfoSize + receiverSideInfoSize));
}

void moeAllToAll(FusedMoeCommKernelParam params, FusedMoeWorkspace workspace, hipStream_t stream)
{
    bool hasBasicFields = params.sendFieldInfo.tokenSelectedSlots != nullptr;
    int warpSendShmSize = params.sendCommMeta.getSingleShmSize();
    int warpRecvShmSize = params.recvCommMeta.getSingleShmSize();
    int warpShmSize = warpSendShmSize;
    int epSize = params.worldInfo.epInfo.epSize;
    TLLM_CHECK_WITH_INFO(warpSendShmSize == warpRecvShmSize, "warpSendShmSize(%d) not same as warpRecvShmSize(%d)",
        warpSendShmSize, warpRecvShmSize);
    int maxGroupCountPerCta = std::min(params.worldInfo.epInfo.epSize, FusedMoeCommunicator::MAX_GROUP_COUNT_PER_BLOCK);
    static int maxDynamicShmSize = fused_moe_impl::computeMoeAlltoallMaxDynamicSharedMemorySize();
    int groupCountPerCta = std::min(maxGroupCountPerCta, maxDynamicShmSize / warpShmSize);

    int maxFieldCount = std::max(params.sendFieldInfo.fieldCount, params.recvFieldInfo.fieldCount);
    TLLM_CHECK_WITH_INFO(params.isLowPrecision == false || maxFieldCount == 1, "low precision only support 1 field");

    auto getFunc = [](int fieldCount, bool lowPrecision)
    {
        switch (fieldCount)
        {
        case 1:
            if (lowPrecision)
                return fused_moe_impl::moeAllToAllKernel<1, true>;
            else
                return fused_moe_impl::moeAllToAllKernel<1>;
        case 2: return fused_moe_impl::moeAllToAllKernel<2>;
        case 3: return fused_moe_impl::moeAllToAllKernel<3>;
        case 4: return fused_moe_impl::moeAllToAllKernel<4>;
        case 5: return fused_moe_impl::moeAllToAllKernel<5>;
        case 6: return fused_moe_impl::moeAllToAllKernel<6>;
        case 7: return fused_moe_impl::moeAllToAllKernel<7>;
        case 8: return fused_moe_impl::moeAllToAllKernel<8>;
        default: return fused_moe_impl::moeAllToAllKernel<8>;
        }
        return fused_moe_impl::moeAllToAllKernel<8>;
    };
    auto* kernelFn = getFunc(maxFieldCount, params.isLowPrecision);

    if (groupCountPerCta * warpShmSize > 48 * 1024)
    {
        TLLM_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
            kernelFn), hipFuncAttributeMaxDynamicSharedMemorySize, groupCountPerCta * warpShmSize));
    }
    for (; groupCountPerCta > 0; groupCountPerCta--)
    {
        int dynamicShmSize = groupCountPerCta * warpShmSize;
        int numBlocks = 0;
        if (hipOccupancyMaxActiveBlocksPerMultiprocessor(
                &numBlocks, kernelFn, WARP_SIZE * groupCountPerCta, dynamicShmSize)
            != hipSuccess)
        {
            continue;
        }
        if (numBlocks >= 1)
        {
            break;
        }
    }
    TLLM_CHECK_WITH_INFO(
        groupCountPerCta >= 1, "computed groupCount=%d, warpShmSize=%d", groupCountPerCta, warpShmSize);
    int ctaPerChannel = (epSize + groupCountPerCta - 1) / groupCountPerCta;
    groupCountPerCta = (epSize + ctaPerChannel - 1) / ctaPerChannel;
    int totalDynamicShmSize = warpShmSize * groupCountPerCta;

    dim3 block = FusedMoeCommunicator::getLaunchBlockDim(groupCountPerCta);
    dim3 grid = FusedMoeCommunicator::getLaunchGridDim(params.worldInfo.epInfo.epSize, groupCountPerCta);
    kernelFn<<<grid, block, totalDynamicShmSize, stream>>>(params, workspace, hasBasicFields);
    TLLM_CUDA_CHECK(hipGetLastError());
}

int FusedMoeCommunicator::maxSmCount = -1;
bool FusedMoeCommunicator::maxSmCountUsed = false;

void setMaxUsableSmCount(int smCount)
{
    FusedMoeCommunicator::setMaxUsableSmCount(smCount);
}

size_t getFusedMoeCommWorkspaceSize(int epSize)
{
    int channelCount = FusedMoeCommunicator::getMoeCommChannelCount(epSize);
    size_t workspaceSize = FusedMoeWorkspace::computeWorkspaceSizePreRank(epSize, channelCount);
    return workspaceSize;
}

void constructWorkspace(FusedMoeWorkspace* workspace, uint64_t* workspacePtr, size_t rankStrideInU64, int epSize)
{
    workspace->workspacePtr = workspacePtr;
    workspace->rankStrideInU64 = rankStrideInU64;
    workspace->channelCount = FusedMoeCommunicator::getMoeCommChannelCount(epSize);
}

void initializeFusedMoeLocalWorkspace(FusedMoeWorkspace* workspace, FusedMoeWorldInfo const& worldInfo)
{
    workspace->initializeLocalWorkspace(worldInfo);
}

namespace fused_moe_comm_tests
{

__global__ void g2sKernel(FusedMoeFieldInfo allFieldInfo, MoeExpertParallelInfo expertParallelInfo,
    MoeSingleCommMeta singleCommMeta, int tokenCount, int* shmDump, bool hasBasicFields)
{
    __shared__ uint64_t allWarpSmemBar[32];
    extern __shared__ int4 allWarpShm[];
    int laneId = threadIdx.x % WARP_SIZE;
    int warpId = threadIdx.x / WARP_SIZE;
    int warpCount = blockDim.x / WARP_SIZE;
    int tokenIndex = warpId + blockIdx.x * warpCount;
    if (tokenIndex >= tokenCount)
    {
        return;
    }

    int singleShmSize = singleCommMeta.singleUncompactAlignedSize;

    tensorrt_llm::kernels::fused_moe_impl::initSmemBar(&allWarpSmemBar[warpId], laneId);
    uint32_t phaseParity = 0;

    uint8_t* sharedMemoryBase = reinterpret_cast<uint8_t*>(allWarpShm) + singleShmSize * warpId;

    if (hasBasicFields)
    {
        tensorrt_llm::kernels::fused_moe_impl::g2sAllFields<true>(
            allFieldInfo, expertParallelInfo, tokenIndex, sharedMemoryBase, warpId, laneId, &allWarpSmemBar[warpId]);
        tensorrt_llm::kernels::fused_moe_impl::waitG2SAllFields<true>(&allWarpSmemBar[warpId], &phaseParity);
    }
    else
    {
        tensorrt_llm::kernels::fused_moe_impl::g2sAllFields<false>(
            allFieldInfo, expertParallelInfo, tokenIndex, sharedMemoryBase, warpId, laneId, &allWarpSmemBar[warpId]);
        tensorrt_llm::kernels::fused_moe_impl::waitG2SAllFields<false>(&allWarpSmemBar[warpId], &phaseParity);
    }

    for (int offset = laneId; offset < singleShmSize / sizeof(int); offset += WARP_SIZE)
    {
        shmDump[tokenIndex * singleShmSize / sizeof(int) + offset] = reinterpret_cast<int*>(sharedMemoryBase)[offset];
    }
}

void launchSingleG2S(FusedMoeFieldInfo const& sendFieldInfo, MoeExpertParallelInfo const& expertParallelInfo,
    int tokenCount, int* shmDump, int warpsPerBlock, bool hasBasicFields, hipStream_t stream)
{
    int warpShmSize = sendFieldInfo.computeSingleUncompactSize(
        expertParallelInfo.topK, sendFieldInfo.expertScales != nullptr, hasBasicFields);
    dim3 blockDim(WARP_SIZE * warpsPerBlock, 1, 1);
    dim3 gridDim((tokenCount + warpsPerBlock - 1) / warpsPerBlock, 1, 1);
    MoeSingleCommMeta singleCommMeta;
    sendFieldInfo.fillMetaInfo(
        &singleCommMeta, expertParallelInfo.topK, sendFieldInfo.expertScales != nullptr, hasBasicFields, false);
    TLLM_CUDA_CHECK(
        hipFuncSetAttribute(reinterpret_cast<const void*>(g2sKernel), hipFuncAttributeMaxDynamicSharedMemorySize, warpShmSize * warpsPerBlock));
    g2sKernel<<<gridDim, blockDim, warpShmSize * warpsPerBlock, stream>>>(
        sendFieldInfo, expertParallelInfo, singleCommMeta, tokenCount, shmDump, hasBasicFields);
    TLLM_CUDA_CHECK(hipGetLastError());
}

__global__ void s2gKernel(FusedMoeFieldInfo recvFieldInfo, MoeExpertParallelInfo expertParallelInfo,
    MoeSingleCommMeta singleCommMeta, int tokenCount, int* shmPreload, bool hasBasicFields)
{
    extern __shared__ int4 allWarpShm[];
    int laneId = threadIdx.x % WARP_SIZE;
    int warpId = threadIdx.x / WARP_SIZE;
    int warpCount = blockDim.x / WARP_SIZE;
    int tokenIndex = warpId + blockIdx.x * warpCount;
    if (tokenIndex >= tokenCount)
    {
        return;
    }
    int singleShmSize = singleCommMeta.singleUncompactAlignedSize;
    uint8_t* sharedMemoryBase = reinterpret_cast<uint8_t*>(allWarpShm) + singleShmSize * warpId;

    for (int offset = laneId; offset < singleShmSize / sizeof(int); offset += WARP_SIZE)
    {
        reinterpret_cast<int*>(sharedMemoryBase)[offset]
            = shmPreload[tokenIndex * singleShmSize / sizeof(int) + offset];
    }
    __syncwarp();

    if (hasBasicFields)
    {
        tensorrt_llm::kernels::fused_moe_impl::s2gAllFields<true>(
            recvFieldInfo, expertParallelInfo, tokenIndex, sharedMemoryBase, warpId, laneId);
    }
    else
    {
        tensorrt_llm::kernels::fused_moe_impl::s2gAllFields<false>(
            recvFieldInfo, expertParallelInfo, tokenIndex, sharedMemoryBase, warpId, laneId);
    }

    tensorrt_llm::kernels::fused_moe_impl::waitS2GBulkRead();
}

void launchSingleS2G(FusedMoeFieldInfo const& recvFieldInfo, MoeExpertParallelInfo const& expertParallelInfo,
    int tokenCount, int* shmPreload, int warpsPerBlock, bool hasBasicFields, hipStream_t stream)
{
    int warpShmSize = recvFieldInfo.computeSingleUncompactSize(
        expertParallelInfo.topK, recvFieldInfo.expertScales != nullptr, hasBasicFields);
    dim3 blockDim(WARP_SIZE * warpsPerBlock, 1, 1);
    dim3 gridDim((tokenCount + warpsPerBlock - 1) / warpsPerBlock, 1, 1);
    MoeSingleCommMeta singleCommMeta;
    recvFieldInfo.fillMetaInfo(
        &singleCommMeta, expertParallelInfo.topK, recvFieldInfo.expertScales != nullptr, hasBasicFields, false);
    TLLM_CUDA_CHECK(
        hipFuncSetAttribute(reinterpret_cast<const void*>(s2gKernel), hipFuncAttributeMaxDynamicSharedMemorySize, warpShmSize * warpsPerBlock));
    s2gKernel<<<gridDim, blockDim, warpShmSize * warpsPerBlock, stream>>>(
        recvFieldInfo, expertParallelInfo, singleCommMeta, tokenCount, shmPreload, hasBasicFields);
    TLLM_CUDA_CHECK(hipGetLastError());
}

__global__ void loopbackKernel(FusedMoeFieldInfo sendFieldInfo, FusedMoeFieldInfo recvFieldInfo,
    MoeExpertParallelInfo expertParallelInfo, MoeSingleCommMeta sendCommMeta, MoeSingleCommMeta recvCommMeta,
    int* recvIndexMapping, int tokenCount, bool hasBasicFields)
{
    __shared__ uint64_t allWarpSmemBar[32];
    extern __shared__ int4 allWarpShm[];
    int laneId = threadIdx.x % WARP_SIZE;
    int warpId = threadIdx.x / WARP_SIZE;
    int warpCount = blockDim.x / WARP_SIZE;
    int tokenIndex = warpId + blockIdx.x * warpCount;
    if (tokenIndex >= tokenCount)
    {
        return;
    }

    int recvTokenIndex = recvIndexMapping[tokenIndex];

    tensorrt_llm::kernels::fused_moe_impl::initSmemBar(&allWarpSmemBar[warpId], laneId);
    uint32_t phaseParity = 0;

    int singleShmSize = sendCommMeta.getSingleShmSize();

    uint8_t* sharedMemoryBase = reinterpret_cast<uint8_t*>(allWarpShm) + singleShmSize * warpId;

    if (hasBasicFields)
    {
        tensorrt_llm::kernels::fused_moe_impl::g2sAllFields<true>(
            sendFieldInfo, expertParallelInfo, tokenIndex, sharedMemoryBase, warpId, laneId, &allWarpSmemBar[warpId]);
    }
    else
    {
        tensorrt_llm::kernels::fused_moe_impl::g2sAllFields<false>(
            sendFieldInfo, expertParallelInfo, tokenIndex, sharedMemoryBase, warpId, laneId, &allWarpSmemBar[warpId]);
    }

    if (hasBasicFields)
    {
        tensorrt_llm::kernels::fused_moe_impl::waitG2SAllFields<true>(&allWarpSmemBar[warpId], &phaseParity);
    }
    else
    {
        tensorrt_llm::kernels::fused_moe_impl::waitG2SAllFields<false>(&allWarpSmemBar[warpId], &phaseParity);
    }

    tensorrt_llm::kernels::fused_moe_impl::packAllFields(sendFieldInfo, tokenIndex, sharedMemoryBase, laneId);

    tokenIndex = recvTokenIndex; // switch to recvTokenIndex;

    tensorrt_llm::kernels::fused_moe_impl::unpackAllFields(recvFieldInfo, tokenIndex, sharedMemoryBase, laneId);

    if (hasBasicFields)
    {
        tensorrt_llm::kernels::fused_moe_impl::s2gAllFields<true>(
            recvFieldInfo, expertParallelInfo, tokenIndex, sharedMemoryBase, warpId, laneId);
    }
    else
    {
        tensorrt_llm::kernels::fused_moe_impl::s2gAllFields<false>(
            recvFieldInfo, expertParallelInfo, tokenIndex, sharedMemoryBase, warpId, laneId);
    }

    cp_async_bulk_wait_group_read<0>();
    __syncwarp();
}

// G2S -> Pack -> Unpack -> S2G
void launchLoopback(FusedMoeFieldInfo const& sendFieldInfo, FusedMoeFieldInfo const& recvFieldInfo,
    MoeExpertParallelInfo const& expertParallelInfo, int* recvIndexMapping, int tokenCount, int warpsPerBlock,
    bool hasBasicFields, hipStream_t stream)
{
    MoeSingleCommMeta sendCommMeta, recvCommMeta;
    sendFieldInfo.fillMetaInfo(
        &sendCommMeta, expertParallelInfo.topK, sendFieldInfo.expertScales != nullptr, hasBasicFields, false);
    recvFieldInfo.fillMetaInfo(
        &recvCommMeta, expertParallelInfo.topK, recvFieldInfo.expertScales != nullptr, hasBasicFields, false);
    int warpSendShmSize = sendCommMeta.getSingleShmSize();
    int warpRecvShmSize = recvCommMeta.getSingleShmSize();
    int warpShmSize = warpSendShmSize;
    TLLM_CHECK_WITH_INFO(warpSendShmSize == warpRecvShmSize, "warpSendShmSize(%d) not same as warpRecvShmSize(%d)",
        warpSendShmSize, warpRecvShmSize);
    dim3 blockDim(WARP_SIZE * warpsPerBlock, 1, 1);
    dim3 gridDim((tokenCount + warpsPerBlock - 1) / warpsPerBlock, 1, 1);
    TLLM_CUDA_CHECK(
        hipFuncSetAttribute(reinterpret_cast<const void*>(loopbackKernel), hipFuncAttributeMaxDynamicSharedMemorySize, warpShmSize * warpsPerBlock));
    loopbackKernel<<<gridDim, blockDim, warpShmSize * warpsPerBlock, stream>>>(sendFieldInfo, recvFieldInfo,
        expertParallelInfo, sendCommMeta, recvCommMeta, recvIndexMapping, tokenCount, hasBasicFields);
    TLLM_CUDA_CHECK(hipGetLastError());
}

template <bool HAS_BASIC_FIELD = true>
__global__ void localFifoSendRecvKernel(FusedMoeFieldInfo sendFieldInfo, FusedMoeFieldInfo recvFieldInfo,
    MoeExpertParallelInfo expertParallelInfo, MoeSingleCommMeta sendCommMeta, MoeSingleCommMeta recvCommMeta,
    FusedMoeWorkspace fusedMoeWorkspace, int* sendIndexMapping, int* recvIndexMapping, int tokenCount)
{
    __shared__ uint64_t allWarpSmemBar[32];
    extern __shared__ int4 allWarpShm[];

    FusedMoeWorldInfo worldInfo;
    worldInfo.epInfo.epRank = 0;
    worldInfo.epInfo.epSize = 1;

    int warpId = threadIdx.x / WARP_SIZE;
    int warpCount = blockDim.x / WARP_SIZE;

    FusedMoePairInfo pairInfo;
    pairInfo.senderRank = 0;
    pairInfo.receiverRank = 0;
    pairInfo.channel = blockIdx.z * warpCount + warpId;
    pairInfo.runChannelCount = gridDim.z * warpCount;

    if (blockIdx.y == 0)
    {
        tensorrt_llm::kernels::fused_moe_impl::SingleChannelCommunicator<MOE_COMM_FIELD_MAX_COUNT, HAS_BASIC_FIELD>
            senderComm(sendFieldInfo, expertParallelInfo, sendCommMeta, fusedMoeWorkspace, worldInfo, pairInfo,
                &allWarpSmemBar[warpId],
                reinterpret_cast<uint8_t*>(&allWarpShm[0]) + warpId * sendCommMeta.getSingleShmSize());
        senderComm.doSend(tokenCount, sendIndexMapping);
    }
    else
    {
        tensorrt_llm::kernels::fused_moe_impl::SingleChannelCommunicator<MOE_COMM_FIELD_MAX_COUNT, HAS_BASIC_FIELD>
            recverComm(recvFieldInfo, expertParallelInfo, recvCommMeta, fusedMoeWorkspace, worldInfo, pairInfo,
                &allWarpSmemBar[warpId],
                reinterpret_cast<uint8_t*>(&allWarpShm[0]) + warpId * recvCommMeta.getSingleShmSize());
        recverComm.doReceive(tokenCount, recvIndexMapping);
    }
}

void launchLocalFifoSendRecv(FusedMoeFieldInfo const& sendFieldInfo, FusedMoeFieldInfo const& recvFieldInfo,
    MoeExpertParallelInfo const& expertParallelInfo, int* sendIndexMapping, int* recvIndexMapping,
    FusedMoeWorkspace fusedMoeWorkspace, int tokenCount, int warpsPerBlock, int blockChannelCount, bool hasBasicFields,
    hipStream_t stream)
{
    MoeSingleCommMeta sendCommMeta, recvCommMeta;
    sendFieldInfo.fillMetaInfo(
        &sendCommMeta, expertParallelInfo.topK, sendFieldInfo.expertScales != nullptr, hasBasicFields, false);
    recvFieldInfo.fillMetaInfo(
        &recvCommMeta, expertParallelInfo.topK, recvFieldInfo.expertScales != nullptr, hasBasicFields, false);
    int warpSendShmSize = sendCommMeta.getSingleShmSize();
    int warpRecvShmSize = recvCommMeta.getSingleShmSize();
    int warpShmSize = warpSendShmSize;
    TLLM_CHECK_WITH_INFO(warpSendShmSize == warpRecvShmSize, "warpSendShmSize(%d) not same as warpRecvShmSize(%d)",
        warpSendShmSize, warpRecvShmSize);
    dim3 blockDim(WARP_SIZE * warpsPerBlock, 1, 1);
    dim3 gridDim(1, 2, blockChannelCount);
    auto* kernelFn = localFifoSendRecvKernel<>;
    if (hasBasicFields)
    {
        kernelFn = localFifoSendRecvKernel<true>;
    }
    else
    {
        kernelFn = localFifoSendRecvKernel<false>;
    }
    TLLM_CUDA_CHECK(
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernelFn), hipFuncAttributeMaxDynamicSharedMemorySize, warpShmSize * warpsPerBlock));
    kernelFn<<<gridDim, blockDim, warpShmSize * warpsPerBlock, stream>>>(sendFieldInfo, recvFieldInfo,
        expertParallelInfo, sendCommMeta, recvCommMeta, fusedMoeWorkspace, sendIndexMapping, recvIndexMapping,
        tokenCount);
    TLLM_CUDA_CHECK(hipGetLastError());
}

} // namespace fused_moe_comm_tests

} // namespace kernels
} // namespace tensorrt_llm
