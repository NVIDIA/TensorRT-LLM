#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif
#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/kernels/samplingTopPKernels.h"
#include <cuda/atomic>

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

#include <cuda/std/limits>
#include <hip/hip_fp16.h>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

using IdxT = int;
using AccT = float;

template <typename T, typename IdxT, typename AccT>
struct alignas(128) Counter
{
    // Address for input value and index
    T const* in;
    IdxT const* inIdx;

    // The original length of the input
    IdxT oriLen;

    // We are processing the values in multiple passes, from most significant to least
    // significant. In each pass, we keep the length of input (`len`) and the `sum` of
    // current pass, and update them at the end of the pass.
    AccT sum;
    IdxT len;
    float p;

    //  `previousLen` is the length of input in previous pass. Note that `previousLen`
    //  rather than `len` is used for the filtering step because filtering is indeed for
    //  previous pass.
    IdxT previousLen;

    // We determine the bits of the k_th value inside the mask processed by the pass. The
    // already known bits are stored in `kthValueBits`. It's used to discriminate a
    // element is a result (written to `out`), a candidate for next pass (written to
    // `outBuf`), or not useful (discarded). The bits that are not yet processed do not
    // matter for this purpose.
    typename hipcub::Traits<T>::UnsignedBits kthValueBits;

    // Record how many elements have passed filtering. It's used to determine the position
    // in the `outBuf` where an element should be written.
    alignas(128) IdxT filterCnt;

    // For a row inside a batch, we may launch multiple thread blocks. This counter is
    // used to determine if the current block is the last running block.
    alignas(128) uint32_t finishedBlockCnt;
};

/*******************************Functions*********************************/
using WideT = float4;

//! \brief Provide a ceiling division operation ie. ceil(a / b)
//! \tparam IntType supposed to be only integers for now!
template <typename IntType>
constexpr __host__ __device__ IntType ceilDiv(IntType a, IntType b)
{
    return (a + b - 1) / b;
}

//! \brief Provide an alignment function ie. ceil(a / b) * b
//! \tparam IntType supposed to be only integers for now!
template <typename IntType>
constexpr __host__ __device__ IntType alignTo(IntType a, IntType b)
{
    return ceilDiv(a, b) * b;
}

//! \brief Calculate the number of buckets based on the number of bits per pass.
//! \tparam BitsPerPass. If BitsPerPass==11, the number of buckets is 2048. If BitsPerPass==8, the number of buckets is
//! 256.
template <int BitsPerPass>
__host__ __device__ int constexpr calcNumBuckets()
{
    return 1 << BitsPerPass;
}

//! \brief Calculate the number of passes based on the number of bits per pass.
//! \tparam BitsPerPass. If BitsPerPass==11, the number of passes is 3. If BitsPerPass==8, the number of passes is 4.
template <typename T, int BitsPerPass>
__host__ __device__ int constexpr calcNumPasses()
{
    return ceilDiv<int>(sizeof(T) * 8, BitsPerPass);
}

/**
 * This implementation processes input from the most to the least significant bit (Bit 0 is the least
 * significant (rightmost)). This way, we can skip some passes in the end at the cost of having an unsorted output.
 */
template <typename T, int BitsPerPass>
__device__ int constexpr calcStartBit(int pass)
{
    int startBit = static_cast<int>(sizeof(T) * 8) - (pass + 1) * BitsPerPass;
    if (startBit < 0)
    {
        startBit = 0;
    }
    return startBit;
}

template <typename T, int BitsPerPass>
__device__ uint32_t constexpr calcMask(int pass)
{
    static_assert(BitsPerPass <= 31);
    int numBits = calcStartBit<T, BitsPerPass>(pass - 1) - calcStartBit<T, BitsPerPass>(pass);
    return (1 << numBits) - 1;
}

template <typename T>
__device__ constexpr uint32_t getNumTotalMantissa()
{
    if constexpr (std::is_same_v<T, half>)
    {
        return 10;
    }
    else if constexpr (std::is_same_v<T, float>)
    {
        return 23;
    }
}

template <typename T>
__device__ uint32_t calcMantissa(T value);

template <>
__device__ uint32_t calcMantissa(float value)
{
    union
    {
        uint32_t bits;
        float value;
    } input;

    input.value = value;

    constexpr uint32_t numTotalMantissa = getNumTotalMantissa<float>();
    uint32_t mask = (1u << numTotalMantissa) - 1;
    return input.bits & mask;
}

__device__ uint32_t calcMantissa(half value)
{
    union
    {
        uint16_t bits;
        half value;
    } input;

    input.value = value;

    constexpr uint32_t numTotalMantissa = getNumTotalMantissa<half>();
    uint32_t t = 0u | input.bits;
    uint32_t mask = (1u << numTotalMantissa) - 1;
    return t & mask;
}

template <typename T>
__device__ uint32_t calcExponent(T value);

template <>
__device__ uint32_t calcExponent(float value)
{
    union
    {
        uint32_t bits;
        float value;
    } input;

    input.value = value;

    constexpr uint32_t numTotalMantissa = getNumTotalMantissa<float>();
    uint32_t mask = (1u << numTotalMantissa) - 1;
    return input.bits & ~mask;
}

template <>
__device__ uint32_t calcExponent(half value)
{
    union
    {
        uint16_t bits;
        half value;
    } input;

    input.value = value;

    constexpr uint32_t numTotalMantissa = getNumTotalMantissa<half>();
    uint32_t t = 0u | input.bits;
    uint32_t mask = (1u << numTotalMantissa) - 1;
    return t & ~mask;
}

__device__ float calcHalfValue(uint32_t count, uint32_t exponent, uint32_t sign, uint64_t bitSum)
{
    constexpr uint32_t numTotalBits = 64; // The bit number of uint64_t
    constexpr uint32_t numOffset = 16;    // The bits number difference between float and half data type
    constexpr uint32_t numTotalMantissaHalf
        = getNumTotalMantissa<half>();    // The bit number of mantissa for half data type
    constexpr uint32_t numTotalMantissaFloat
        = getNumTotalMantissa<float>();   // The bit number of mantissa for float data type

    uint64_t extraInMatissa = (bitSum >> numTotalMantissaHalf);

    // Count the bit number for exceeding mantissa and the extra unwritten 1s
    uint32_t numExtra = 0;
    uint32_t numDeNorm = 0;
    int numNorm = 0;
    uint32_t mask = 0;
    extraInMatissa = (exponent == 0) ? extraInMatissa : extraInMatissa + count;
    numExtra = numTotalBits - __clzll(extraInMatissa);
    numNorm = (exponent == 0) ? 0 : -1;
    if (extraInMatissa == 0)
    {
        numDeNorm = numTotalMantissaHalf - (numTotalBits - __clzll(bitSum));
    }
    exponent = exponent + ((numExtra + numNorm + 127 - 15 - numDeNorm) << numTotalMantissaHalf);
    // As extra bits (extraInMatissa) need to be part of the mantissa, we have to move the current
    // mantissa within the range of [0-23]bits.
    // This is the only step cause precision loss
    uint32_t mantissa;
    if (extraInMatissa != 0)
    {
        int numMove = numTotalMantissaFloat - (numExtra - 1);
        mask = (1u << (numExtra - 1)) - 1;
        // As the first bit of extraInMatissa is the unwritten 1,
        // we need to mask that to zero
        extraInMatissa = extraInMatissa & mask;
        if (numMove > 0)
        {
            extraInMatissa = extraInMatissa << numMove;
            mask = (1u << numTotalMantissaHalf) - 1;
            mantissa = (((bitSum & mask) << (numTotalMantissaFloat - numTotalMantissaHalf)) >> (numExtra - 1))
                | extraInMatissa;
        }
        else
        {
            mantissa = extraInMatissa >> (-1 * numMove);
        }
    }
    else
    {
        mask = (1u << numTotalMantissaHalf) - 1;
        mantissa = bitSum << (numDeNorm + 1);
        mantissa = mantissa & mask;
        mantissa = mantissa << (numTotalMantissaFloat - numTotalMantissaHalf);
    }

    uint32_t bitFloat = (sign << numOffset) | (exponent << (numTotalMantissaFloat - numTotalMantissaHalf)) | mantissa;
    return reinterpret_cast<float&>(bitFloat);
}

__device__ float calcFloatValue(uint32_t count, uint32_t exponent, uint64_t bitSum)
{
    constexpr uint32_t numTotalBits = 64;
    constexpr uint32_t numTotalMantissa = getNumTotalMantissa<float>();
    uint64_t extraInMatissa = (bitSum >> numTotalMantissa);
    // Count the bit number for exceeding mantissa and the extra unwritten 1s
    uint32_t numExtra;
    int numNorm = 0;
    uint32_t mask = 0;
    extraInMatissa = (exponent == 0) ? extraInMatissa : extraInMatissa + count;
    numExtra = numTotalBits - __clzll(extraInMatissa);
    numNorm = (exponent == 0) ? 0 : -1;
    exponent = exponent + ((numExtra + numNorm) << numTotalMantissa);
    // As extra integers need to be part of the mantissa, we have to move the current
    // mantissa within the range of [0-23]bits.
    // This is the only step cause precision loss
    uint32_t mantissa;
    if (extraInMatissa != 0)
    {
        int numMove = numTotalMantissa - (numExtra - 1);
        // As the first bit of extraInMatissa is the unwritten 1,
        // we need to mask that to zero
        mask = (1u << (numExtra - 1)) - 1;
        extraInMatissa = extraInMatissa & mask;
        if (numMove > 0)
        {
            extraInMatissa = extraInMatissa << numMove;
            mask = (1u << numTotalMantissa) - 1;
            mantissa = ((bitSum & mask) >> (numExtra - 1)) | extraInMatissa;
        }
        else
        {
            mantissa = extraInMatissa >> (-1 * numMove);
        }
    }
    else
    {
        mantissa = bitSum;
    }
    uint32_t bitFloat = exponent | mantissa;
    return reinterpret_cast<float&>(bitFloat);
}

template <typename T, typename HisT, bool isDeterministic = false>
__device__ constexpr void calcAtomicAdd(HisT* dst, T value)
{
    if constexpr (isDeterministic)
    {
        uint32_t mantissa = calcMantissa(value);
        if constexpr (std::is_same_v<T, half>)
        {
            atomicAdd(dst, mantissa);
        }
        else
        {
            // Have to use reinterpret_cast() to convert uint64_t to "unsigned long long"
            // Otherwise, the complication will report the follow error:
            //"error: no instance of overloaded function "atomicAdd" matches the argument list
            // argument types are: (uint64_t *, uint64_t)"
            atomicAdd(reinterpret_cast<unsigned long long*>(dst), static_cast<HisT>(mantissa));
        }
    }
    else
    {
        if constexpr (std::is_same_v<T, half>)
        {
            atomicAdd(dst, __half2float(value));
        }
        else
        {
            atomicAdd(dst, value);
        }
    }
}

/**
 * Use CUB to twiddle bits.
 */
template <typename T>
__device__ typename hipcub::Traits<T>::UnsignedBits twiddleIn(T key, bool selectMin)
{
    auto bits = reinterpret_cast<typename hipcub::Traits<T>::UnsignedBits&>(key);
    bits = hipcub::Traits<T>::TwiddleIn(bits);
    if (!selectMin)
    {
        bits = ~bits;
    }
    return bits;
}

template <typename T>
__device__ T twiddleOut(typename hipcub::Traits<T>::UnsignedBits bits, bool selectMin)
{
    if (!selectMin)
    {
        bits = ~bits;
    }
    bits = hipcub::Traits<T>::TwiddleOut(bits);
    return reinterpret_cast<T&>(bits);
}

/**
 * Find the bucket based on the radix
 */
template <typename T, int BitsPerPass>
__device__ int calcBucket(T x, int startBit, uint32_t mask, bool selectMin)
{
    static_assert(BitsPerPass <= sizeof(int) * 8 - 1, "BitsPerPass is too large that the result type could not be int");
    return (twiddleIn(x, selectMin) >> startBit) & mask;
}

/**
 * This function calculate the bufLen, which is the size of buffer.
 * When the number of candidates for next pass exceeds the bufLen, we choose not to store the candidates. Otherwise, we
 * will load candidates from the original input data.
 */
template <typename T, typename IdxT>
__host__ __device__ IdxT calcBufLen(IdxT len)
{
    // This ratio is calculated based on the element number.
    // If we choose to write the buffers, it means (sizeof(T)+sizeof(IdxT))*bufLen bytes of storing and loading.
    // To ensure we do not access more than len*sizeof(T) bytes. bufLen should be smaller than:
    // len*sizeof(T)/2*(sizeof(T) + sizeof(IdxT)) = len/(2 + sizeof(IdxT) * 2 / sizeof(T))).
    IdxT constexpr ratio = 2 + sizeof(IdxT) * 2 / sizeof(T);
    // Even such estimation is too conservative (due to the global coalescing access). So based on our experiments, we
    // further decrease bufLen by 1/8
    IdxT bufLen = len / (ratio * 8);

    // Align the address to 256 bytes
    bufLen = alignTo(bufLen, 256);
    return bufLen;
}

/**
 * Use ping-pong buffer and set the inBuf and outBuf based on the pass value.
 */
template <typename T, typename IdxT>
__host__ __device__ void setBufPointers(T const* in, IdxT const* inIdx, T* buf1, IdxT* idxBuf1, T* buf2, IdxT* idxBuf2,
    int pass, T const*& inBuf, IdxT const*& inIdxBuf, T*& outBuf, IdxT*& outIdxBuf)
{
    if (pass == 0)
    {
        inBuf = in;
        inIdxBuf = nullptr;
        outBuf = nullptr;
        outIdxBuf = nullptr;
    }
    else if (pass == 1)
    {
        inBuf = in;
        inIdxBuf = inIdx;
        outBuf = buf1;
        outIdxBuf = idxBuf1;
    }
    else if (pass % 2 == 0)
    {
        inBuf = buf1;
        inIdxBuf = idxBuf1;
        outBuf = buf2;
        outIdxBuf = idxBuf2;
    }
    else
    {
        inBuf = buf2;
        inIdxBuf = idxBuf2;
        outBuf = buf1;
        outIdxBuf = idxBuf1;
    }
}

//! \brief Map a Func over the input data, using vectorized load instructions if possible.
//! \tparam T element type
//! \tparam IdxT indexing type
//! \tparam Func void (T x, IdxT idx)
//! \param threadRank rank of the calling thread among all participating threads
//! \param numThreads number of the threads that participate in processing
//! \param in the input data
//! \param len the number of elements to read
//! \param f the lambda taking two arguments (T x, IdxT idx)
template <typename T, typename IdxT, typename Func>
__device__ void vectorizedProcess(size_t threadRank, size_t numThreads, T const* in, IdxT len, Func f)
{
    int constexpr WARP_SIZE = 32;
    if constexpr (sizeof(T) >= sizeof(WideT))
    {
        for (IdxT i = threadRank; i < len; i += numThreads)
        {
            f(in[i], i);
        }
    }
    else
    {
        static_assert(sizeof(WideT) % sizeof(T) == 0);
        int constexpr itemsPerScalar = sizeof(WideT) / sizeof(T);

        // TODO: it's UB
        union
        {
            WideT scalar;
            T array[itemsPerScalar];
        } wide;

        int skipCnt = (reinterpret_cast<size_t>(in) % sizeof(WideT))
            ? ((sizeof(WideT) - reinterpret_cast<size_t>(in) % sizeof(WideT)) / sizeof(T))
            : 0;
        if (skipCnt > len)
        {
            skipCnt = len;
        }
        WideT const* inCast = reinterpret_cast<decltype(inCast)>(in + skipCnt);
        IdxT const lenCast = (len - skipCnt) / itemsPerScalar;

        for (IdxT i = threadRank; i < lenCast; i += numThreads)
        {
            wide.scalar = inCast[i];
            IdxT const real_i = skipCnt + i * itemsPerScalar;
#pragma unroll
            for (int j = 0; j < itemsPerScalar; ++j)
            {
                f(wide.array[j], real_i + j);
            }
        }

        static_assert(WARP_SIZE >= itemsPerScalar);
        // and because itemsPerScalar > skipCnt, WARP_SIZE > skipCnt
        // no need to use loop
        if (threadRank < skipCnt)
        {
            f(in[threadRank], threadRank);
        }
        // because lenCast = (len - skipCnt) / itemsPerScalar,
        // lenCast * itemsPerScalar + itemsPerScalar > len - skipCnt;
        // and so
        // len - (skipCnt + lenCast * itemsPerScalar) < itemsPerScalar <=
        // WARP_SIZE no need to use loop
        IdxT const remain_i = skipCnt + lenCast * itemsPerScalar + threadRank;
        if (remain_i < len)
        {
            f(in[remain_i], remain_i);
        }
    }
}

/**
 * Fused filtering of the current pass and building histogram for the next pass (see steps 4 & 1 in `airTopPSampling`
 * description).
 */
template <typename T, typename IdxT, typename AccT, typename HisT, int BitsPerPass, bool isDeterministic = false>
__device__ __forceinline__ void filterAndHistogram(T const* inBuf, IdxT const* inIdxBuf, T* outBuf, IdxT* outIdxBuf,
    int previousLen, Counter<T, IdxT, AccT>* counter, HisT* histogram, IdxT* countHistogram, HisT* histogramSmem,
    IdxT* countHistogramSmem, int pass, float* outputLogProbs, float* cumLogProbs, IdxT** ids, IdxT const* endIds,
    IdxT* sequenceLengths, FinishedState* finishedOutput, int const batchId, int maxBatchSize, bool earlyStop)
{
    static_assert(std::is_same_v<T, half> | std::is_same_v<T, float>, "T needs to be either half or float");
    static_assert(std::is_same_v<AccT, float>, "AccT needs to be float");

    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    bool constexpr selectMin = false;

    for (IdxT i = threadIdx.x; i < numBuckets; i += blockDim.x)
    {
        histogramSmem[i] = 0;
        countHistogramSmem[i] = 0;
    }
    __syncthreads();

    int const startBit = calcStartBit<T, BitsPerPass>(pass);
    uint32_t const mask = calcMask<T, BitsPerPass>(pass);

    if (pass == 0)
    {
        // Passed to vectorizedProcess, this function executes in all blocks in
        // parallel, i.e. the work is split along the input (both, in batches and
        // chunks of a single row). Later, the histograms are merged using
        // atomicAdd.
        auto f = [selectMin, startBit, mask, histogramSmem, countHistogramSmem](T value, IdxT)
        {
            int bucket = calcBucket<T, BitsPerPass>(value, startBit, mask, selectMin);
            calcAtomicAdd<T, HisT, isDeterministic>(histogramSmem + bucket, value);
            atomicAdd(countHistogramSmem + bucket, static_cast<IdxT>(1));
        };
        vectorizedProcess(static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x,
            static_cast<size_t>(blockDim.x) * gridDim.x, inBuf, previousLen, f);
    }
    else
    {
        IdxT* pFilterCnt = &counter->filterCnt;
        auto const kthValueBits = counter->kthValueBits;
        int const previousStartBit = calcStartBit<T, BitsPerPass>(pass - 1);

        // See the remark above on the distributed execution of `f` using
        // vectorizedProcess.
        auto f = [inIdxBuf, outBuf, outIdxBuf, selectMin, startBit, mask, previousStartBit, kthValueBits, pFilterCnt,
                     histogramSmem, countHistogramSmem, outputLogProbs, cumLogProbs, ids, endIds, sequenceLengths,
                     finishedOutput, batchId, maxBatchSize, earlyStop](T value, IdxT i)
        {
            auto const previousBits = (twiddleIn(value, selectMin) >> previousStartBit) << previousStartBit;
            if (previousBits == kthValueBits)
            {
                if (earlyStop)
                {

                    int const currentStep = sequenceLengths ? sequenceLengths[batchId] : 0;
                    IdxT index = inIdxBuf ? inIdxBuf[i] : i;
                    ids[batchId][currentStep] = index;
                    float valueFloat;
                    if constexpr (std::is_same_v<T, half>)
                    {
                        valueFloat = __half2float(value);
                    }
                    else
                    {
                        valueFloat = value;
                    }
                    epilogue(valueFloat, index, outputLogProbs, cumLogProbs, endIds, sequenceLengths, finishedOutput,
                        batchId, maxBatchSize);
                }
                if (outBuf)
                {
                    IdxT pos = atomicAdd(pFilterCnt, static_cast<IdxT>(1));
                    outBuf[pos] = value;
                    outIdxBuf[pos] = inIdxBuf ? inIdxBuf[i] : i;
                }

                int bucket = calcBucket<T, BitsPerPass>(value, startBit, mask, selectMin);
                calcAtomicAdd<T, HisT, isDeterministic>(histogramSmem + bucket, value);
                atomicAdd(countHistogramSmem + bucket, static_cast<IdxT>(1));
            }
        };
        vectorizedProcess(static_cast<size_t>(blockIdx.x) * blockDim.x + threadIdx.x,
            static_cast<size_t>(blockDim.x) * gridDim.x, inBuf, previousLen, f);
    }

    __syncthreads();
    if (earlyStop)
    {
        return;
    }

    // merge histograms produced by individual blocks
    for (int i = threadIdx.x; i < numBuckets; i += blockDim.x)
    {
        if (histogramSmem[i] != 0)
        {
            if constexpr ((isDeterministic) && (std::is_same_v<T, float>) )
            {
                // Have to use reinterpret_cast() to convert uint64_t to "unsigned long long"
                // Otherwise, the complication will report the follow error:
                //"error: no instance of overloaded function "atomicAdd" matches the argument list
                // argument types are: (uint64_t *, uint64_t)"
                atomicAdd(reinterpret_cast<unsigned long long*>(histogram + i), histogramSmem[i]);
            }
            else
            {
                atomicAdd(histogram + i, histogramSmem[i]);
            }
        }
        if (countHistogramSmem[i] != 0)
        {
            atomicAdd(countHistogram + i, countHistogramSmem[i]);
        }
    }
}

/**
 *  Replace histogram with its own prefix sum (step 2 in `airTopPSampling` description)
 */
template <typename IdxT, int BitsPerPass, int BlockSize>
__device__ void scan(IdxT volatile* histogram, IdxT* histogramOut)
{
    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    if constexpr (numBuckets >= BlockSize)
    {
        static_assert(numBuckets % BlockSize == 0);
        int constexpr itemsPerThread = numBuckets / BlockSize;
        typedef hipcub::BlockLoad<IdxT, BlockSize, itemsPerThread, hipcub::BLOCK_LOAD_TRANSPOSE> BlockLoad;
        typedef hipcub::BlockStore<IdxT, BlockSize, itemsPerThread, hipcub::BLOCK_STORE_TRANSPOSE> BlockStore;
        typedef hipcub::BlockScan<IdxT, BlockSize> BlockScan;

        __shared__ union
        {
            typename BlockLoad::TempStorage load;
            typename BlockScan::TempStorage scan;
            typename BlockStore::TempStorage store;
        } tempStorage;

        IdxT threadData[itemsPerThread];

        BlockLoad(tempStorage.load).Load(histogram, threadData);
        __syncthreads();

        BlockScan(tempStorage.scan).InclusiveSum(threadData, threadData);
        __syncthreads();

        BlockStore(tempStorage.store).Store(histogramOut, threadData);
    }
    else
    {
        typedef hipcub::BlockScan<IdxT, BlockSize> BlockScan;
        __shared__ typename BlockScan::TempStorage tempStorage;

        IdxT threadData = 0;
        if (threadIdx.x < numBuckets)
        {
            threadData = histogram[threadIdx.x];
        }

        BlockScan(tempStorage).InclusiveSum(threadData, threadData);
        __syncthreads();

        if (threadIdx.x < numBuckets)
        {
            histogramOut[threadIdx.x] = threadData;
        }
    }
}

/**
 * Computes sequenceLength, finished state, outputLogProbs, and cumLogProbs.
 */
template <typename T, typename IdxT>
__device__ void epilogue(T const value, IdxT const index, float* outputLogProbs, float* cumLogProbs, IdxT const* endIds,
    IdxT* sequenceLengths, FinishedState* finishedOutput, int const batchId, int maxBatchSize)
{
    if (outputLogProbs != nullptr || cumLogProbs != nullptr)
    {
        float res = logf(value);
        if (outputLogProbs)
        {
            auto const curLen = sequenceLengths ? sequenceLengths[batchId] : 0;
            outputLogProbs[curLen * maxBatchSize + batchId] = res;
        }
        if (cumLogProbs)
        {
            cumLogProbs[batchId] += res;
        }
    }
    if (endIds && index == endIds[batchId])
    {
        if (finishedOutput != nullptr)
        {
            finishedOutput[batchId].setFinishedEOS();
        }
        // Do not increase seq len when EOS is generated. Seq len should always contain only tokens to be outputted
    }
    else if (sequenceLengths != nullptr)
    {
        // We don't need to set output finished state as it is assumed to be in non finished state
        sequenceLengths[batchId] += 1;
    }
}

/**
 *  Find the target element.
 *  (steps 4 in `airTopPSampling` description)
 */
template <typename T, typename IdxT, typename AccT, int BitsPerPass, int BlockSize, bool isDeterministic = false>
__device__ void lastFilter(T const* inBuf, IdxT const* inIdxBuf, IdxT currentLen, Counter<T, IdxT, AccT>* counter,
    float* outputLogProbs, float* cumLogProbs, IdxT** ids, IdxT const* endIds, IdxT* sequenceLengths,
    FinishedState* finishedOutput, int const batchId, int maxBatchSize, IdxT* lastIdxBuf, IdxT* countHistogram)
{
    auto const kthValueBits = counter->kthValueBits;
    auto const equalValue = twiddleOut<T>(kthValueBits, false);
    int const currentStep = sequenceLengths ? sequenceLengths[batchId] : 0;
    IdxT* outIdx = &ids[batchId][currentStep];

    float equalValueFloat;
    if constexpr (std::is_same_v<T, half>)
    {
        equalValueFloat = __half2float(equalValue);
    }
    else
    {
        equalValueFloat = equalValue;
    }
    if constexpr (!isDeterministic)
    {

        for (IdxT i = threadIdx.x; i < currentLen; i += blockDim.x)
        {
            if (inBuf[i] == equalValue)
            {
                *outIdx = inIdxBuf ? inIdxBuf[i] : i;
                break;
            }
        }
    }
    else
    {
        IdxT const bufLen = calcBufLen<T>(counter->oriLen);
        IdxT neededNumOfKth = counter->sum > 0 ? ceil(counter->sum / equalValueFloat) : 1;

        if (counter->len < neededNumOfKth)
        {
            neededNumOfKth = counter->len;
        }

        if (neededNumOfKth < bufLen)
        {
            for (int i = threadIdx.x; i < neededNumOfKth; i += blockDim.x)
            {
                lastIdxBuf[i] = cuda::std::numeric_limits<IdxT>::max();
            }
            __threadfence_block();
            __syncthreads();

            cuda::atomic_ref<IdxT, cuda::thread_scope_block> refLast(lastIdxBuf[neededNumOfKth - 1]);

            for (IdxT i = threadIdx.x; i < currentLen; i += blockDim.x)
            {
                if (inBuf[i] == equalValue)
                {
                    IdxT newIdx = inIdxBuf ? inIdxBuf[i] : i;
                    if (newIdx < refLast.load(cuda::memory_order_relaxed))
                    {
                        for (int j = 0; j < neededNumOfKth; j++)
                        {
                            IdxT preIdx = atomicMin_block(&lastIdxBuf[j], newIdx);
                            if (preIdx > newIdx)
                            {
                                newIdx = preIdx;
                            }
                        }
                    }
                }
            }
            __syncthreads();
            if (threadIdx.x == 0)
            {
                *outIdx = refLast.load(cuda::memory_order_relaxed);
            }
        }
        else
        {
            int numPass = calcNumPasses<IdxT, BitsPerPass>();
            int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
            __shared__ typename hipcub::Traits<IdxT>::UnsignedBits kthValueBitsIdx;
            __shared__ IdxT neededNumOfKthSmem;
            if (threadIdx.x == 0)
            {
                kthValueBitsIdx = 0;
                neededNumOfKthSmem = neededNumOfKth;
            }
            __syncthreads();
            for (int pass = 0; pass < numPass; pass++)
            {
                for (IdxT i = threadIdx.x; i < numBuckets; i += blockDim.x)
                {
                    countHistogram[i] = 0;
                }
                __syncthreads();

                int preNeededNumOfKth = neededNumOfKthSmem;
                int const startBit = calcStartBit<IdxT, BitsPerPass>(pass);
                uint32_t const mask = calcMask<IdxT, BitsPerPass>(pass);
                for (IdxT j = threadIdx.x; j < currentLen; j += blockDim.x)
                {
                    if (inBuf[j] == equalValue)
                    {
                        IdxT newIdx = inIdxBuf ? inIdxBuf[j] : j;
                        bool isQualified = (pass == 0) ? true : false;
                        if (pass > 0)
                        {
                            int const previousStartBit = calcStartBit<IdxT, BitsPerPass>(pass - 1);
                            auto const previousBits = (twiddleIn(newIdx, true) >> previousStartBit) << previousStartBit;
                            if (previousBits == kthValueBitsIdx)
                            {
                                isQualified = true;
                            }
                        }
                        if (isQualified)
                        {
                            int bucket = calcBucket<IdxT, BitsPerPass>(newIdx, startBit, mask, true);
                            atomicAdd(countHistogram + bucket, static_cast<IdxT>(1));
                        }
                    }
                } // end histogram
                __syncthreads();

                scan<IdxT, BitsPerPass, BlockSize>(countHistogram, countHistogram); // prefix sum
                __syncthreads();
                // Locate the bucket
                for (int i = threadIdx.x; i < numBuckets; i += blockDim.x)
                {
                    IdxT prev = (i == 0) ? 0 : countHistogram[i - 1];
                    IdxT cur = countHistogram[i];
                    // one and only one thread will satisfy this condition, so counter is
                    // written by only one thread
                    if (prev < preNeededNumOfKth && preNeededNumOfKth <= cur)
                    {
                        neededNumOfKthSmem = neededNumOfKthSmem - prev;
                        typename hipcub::Traits<IdxT>::UnsignedBits bucket = i;
                        kthValueBitsIdx |= bucket << startBit;
                    }
                }
                __syncthreads();
            }
            if (threadIdx.x == 0)
            {
                *outIdx = twiddleOut<IdxT>(kthValueBitsIdx, true);
            }
        }
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        epilogue(equalValueFloat, *outIdx, outputLogProbs, cumLogProbs, endIds, sequenceLengths, finishedOutput,
            batchId, maxBatchSize);
    }
}

/******************************Kernel**********************************/
/**
 * We call this parallel top-p algorithm AIR Top-P, because this method is based on our previous work called AIR Top-K.
 * Details about AIR Top-K can be found here https://dl.acm.org/doi/10.1145/3581784.360706, the open-source code is here
 * https://github.com/rapidsai/raft/blob/main/cpp/include/raft/matrix/detail/select_radix.cuh
 *
 * It is expected to call this kernel multiple times (passes), in each pass we process a radix,
 * going from the most significant towards the least significant bits (MSD).
 *
 * Conceptually, each pass consists of 4 steps:
 *
 * 1. Calculate histogram
 *      First, transform bits into a digit, the value of which is in the range
 *      [0, 2^{BITS_PER_PASS}-1]. Then count the frequency of each digit value along with the summation of corresponding
 * elements and the result is a countHistogram and histogram. That is, countHistogram[i] contains the count of inputs
 * having value i.
 *
 * 2. Scan the histogram
 *      Inclusive prefix sum is computed for the histogram. After this step, histogram[i] contains
 *      the prefix-sum of inputs having value <= i.
 *
 * 3. Find the bucket j of the histogram that just exceed the p*total_sum value falls into
 *
 * 4. Filtering
 *      Input elements whose digit value <j are the top-p elements. Since the k-th value must be in
 *      the bucket j, we write all elements in bucket j into a intermediate buffer out_buf. For the
 *      next pass, these elements are used as input, and we update the counter->sum accordingly. T
 *
 * In the implementation, the filtering step is delayed to the next pass so the filtering and
 * histogram computation are fused. In this way, inputs are read once rather than twice.
 *
 * During the filtering step, we won't write candidates (elements in bucket j) to `out_buf` if the
 * number of candidates is larger than the length of `out_buf` (this could happen when the leading
 * bits of input values are almost the same). And then in the next pass, inputs are read from `in`
 * rather than from `in_buf`. The benefit is that we can save the cost of writing candidates and
 * their indices.
 */
template <typename T, typename IdxT, typename AccT, typename HisT, int BitsPerPass, int BlockSize,
    bool isFusedFilter = false, bool isDeterministic = false>
__global__ void airTopPSampling(Counter<T, IdxT, AccT>* counters, HisT* histograms, IdxT* countHistograms, IdxT** ids,
    int* sequenceLengths, FinishedState const* finishedInput, FinishedState* finishedOutput, float* cumLogProbs,
    float* outputLogProbs, IdxT const* endIds, int const maxBatchSize, bool const* skipDecode, int const pass, T* buf1,
    IdxT* idxBuf1, T* buf2, IdxT* idxBuf2, int32_t const* batchSlots)
{
    static_assert(std::is_same_v<T, half> | std::is_same_v<T, float>, "T needs to be either half or float");
    static_assert(std::is_same_v<AccT, float>, "AccT needs to be float");

    int const tid = threadIdx.x;
    int const batchId = blockIdx.y;
    auto const batchSlot = batchSlots ? batchSlots[batchId] : batchId;
    auto counter = counters + batchId;

    // Skip kernel if this sampling method is not chosen
    FinishedState const finishState = finishedInput != nullptr ? finishedInput[batchSlot] : FinishedState::empty();
    if ((skipDecode != nullptr && skipDecode[batchSlot]) || (finishState.isSkipDecoding()))
    {
        return;
    }

    // Exit early if sequence has finished
    if (finishState.isFinished())
    {
        if (pass == 0 && tid == 0)
        {
            if (finishedOutput != nullptr)
            {
                finishedOutput[batchSlot] = finishState;
            }
        }
        return;
    }

    /// Set length
    AccT currentSum;
    IdxT previousLen;
    IdxT currentLen;

    if (pass == 0)
    {
        currentSum = 0;
        previousLen = counter->len;
        // Need to do this so setting counter->previousLen for the next pass is correct.
        // This value is meaningless for pass 0, but it's fine because pass 0 won't be the
        // last pass in this implementation so pass 0 won't hit the "if (pass ==
        // numPasses - 1)" branch.
        currentLen = counter->len;
    }
    else
    {
        currentSum = counter->sum;
        currentLen = counter->len;
        previousLen = counter->previousLen;
    }
    if (currentLen == 0)
    {
        return;
    }
    bool const earlyStop = (currentLen == 1);
    IdxT const bufLen = calcBufLen<T>(counter->oriLen);

    /// Set address
    T const* inBuf = nullptr;
    IdxT const* inIdxBuf = nullptr;
    T* outBuf = nullptr;
    IdxT* outIdxBuf = nullptr;

    setBufPointers(counter->in, counter->inIdx, buf1 + bufLen * batchId, idxBuf1 + bufLen * batchId,
        buf2 + bufLen * batchId, idxBuf2 + bufLen * batchId, pass, inBuf, inIdxBuf, outBuf, outIdxBuf);

    // "previousLen > bufLen" means previous pass skips writing buffer
    if (pass == 0 || pass == 1 || previousLen > bufLen)
    {
        inBuf = counter->in;
        inIdxBuf = counter->inIdx;
        previousLen = counter->oriLen;
    }
    // "currentLen > bufLen" means current pass will skip writing buffer
    if (pass == 0 || currentLen > bufLen)
    {
        outBuf = nullptr;
        outIdxBuf = nullptr;
    }
    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    auto histogram = histograms + batchId * numBuckets;
    auto countHistogram = countHistograms + batchId * numBuckets;
    __shared__ HisT histogramSmem[numBuckets];
    __shared__ IdxT countHistogramSmem[numBuckets];
    AccT* histValueSmem = reinterpret_cast<AccT*>(histogramSmem);

    filterAndHistogram<T, IdxT, AccT, HisT, BitsPerPass, isDeterministic>(inBuf, inIdxBuf, outBuf, outIdxBuf,
        previousLen, counter, histogram, countHistogram, histogramSmem, countHistogramSmem, pass, outputLogProbs,
        cumLogProbs, ids, endIds, sequenceLengths, finishedOutput, batchSlot, maxBatchSize, earlyStop);

    __syncthreads();
    __threadfence();

    bool isLastBlock = false;
    if (threadIdx.x == 0)
    {
        uint32_t finished = atomicInc(&counter->finishedBlockCnt, gridDim.x - 1);
        isLastBlock = (finished == (gridDim.x - 1));
    }

    if (__syncthreads_or(isLastBlock))
    {
        if (earlyStop)
        {
            if (threadIdx.x == 0)
            {
                // avoid duplicated epilgue()
                counter->previousLen = 0;
                counter->len = 0;
            }
            return;
        }

        if constexpr (isDeterministic)
        {
            for (int i = threadIdx.x; i < numBuckets; i += blockDim.x)
            {
                uint64_t value = (uint64_t) histogram[i];
                IdxT count = countHistogram[i];

                if (count != 0)
                {
                    uint32_t startBit = calcStartBit<T, BitsPerPass>(pass);
                    [[maybe_unused]] float bucketValueFloat;
                    if constexpr (std::is_same_v<T, half>)
                    {
                        // To acquire the summation in single-precision format, we need to get the original exponent
                        // value first counter->kthValueBits stores the bits selected by previous pass, which contains
                        // the bit corresponds to the exponent value
                        uint16_t bucketValue = counter->kthValueBits;

                        // For the first pass, different bucket indices correspond to different exponents.
                        // The bucket index can be used to deduce the exponent.
                        if (pass == 0)
                        {
                            // Right shift the bucket index with startBit bits (5 bits for half-precision when pass==0),
                            // so that the bucket index fills the bit related to exponent.
                            bucketValue = i << startBit;
                        }
                        uint32_t exponent = calcExponent(twiddleOut<T>(bucketValue, false));
                        uint32_t mask = (1u << (sizeof(half) * CHAR_BIT - 1)) - 1;
                        uint32_t sign = exponent & (~mask);
                        exponent = exponent & mask;
                        float tmp = calcHalfValue((uint32_t) count, exponent, sign, value);
                        histValueSmem[i] = tmp;
                    }
                    else
                    {
                        // To acquire the summation in single-precision format, we need to get the original exponent
                        // value first
                        uint32_t bucketValue = counter->kthValueBits;
                        if (pass == 0)
                        {
                            // Right shift the bucket index with startBit bits (22 bits for single-precision when
                            // pass==0), so that the bucket index fills the bit related to exponent.
                            bucketValue = i << startBit;
                        }
                        bucketValueFloat = twiddleOut<T>(bucketValue, false);
                        uint32_t exponent = calcExponent(bucketValueFloat);
                        histValueSmem[i] = calcFloatValue((uint32_t) count, exponent, value);
                    }
                }
                else
                {
                    histValueSmem[i] = 0.0f;
                }
            }
        }

        // To avoid the error related to the prefix sum from cub, we find the bucket sequentially.
        int constexpr WARP_SIZE = 32;
        int constexpr WARP_COUNT = numBuckets / WARP_SIZE;
        namespace cg = cooperative_groups;
        cg::thread_block block = cg::this_thread_block();
        cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
        AccT* histPtr = isDeterministic ? histValueSmem : reinterpret_cast<AccT*>(histogram);
        __shared__ AccT warpSum[WARP_COUNT];
        __shared__ cuda::atomic<AccT, cuda::thread_scope_block> blockSum;
        if constexpr (BitsPerPass != 11)
        {
            for (int i = threadIdx.x; i < numBuckets; i += BlockSize)
            {
                warpSum[i] = 0;
            }
            __syncthreads();
        }

        // Acquire the summation of each 32 buckets
        for (int i = threadIdx.x; i < numBuckets; i += BlockSize)
        {
            reduce_store_async(warp, warpSum + i / WARP_SIZE, histPtr[i], cg::plus<float>{});
        }
        __syncthreads();

        // Acquire the summation of all the 2048 buckets
        if (threadIdx.x < WARP_SIZE)
        {
            reduce_store_async(warp, blockSum, warpSum[threadIdx.x], cg::plus<float>{});
            if constexpr (BitsPerPass == 11)
            {
                reduce_update_async(warp, blockSum, warpSum[threadIdx.x + WARP_SIZE], cg::plus<float>{});
            }
        }
        __syncthreads();

        // Update currentSum
        if (pass == 0)
        {
            currentSum = blockSum * counter->p;
        }

        if (threadIdx.x == 0)
        {
            AccT prev = 0;

            // Add 32 elements each step
            int iStep = 0;
            int targetStep = 0;
            for (; iStep < WARP_COUNT; iStep++)
            {
                if (warpSum[iStep])
                {
                    targetStep = iStep;
                    if ((prev + warpSum[iStep]) >= currentSum)
                    {
                        break;
                    }
                    prev += warpSum[iStep];
                }
            }

            int targetIdx = 0;
            for (int i = targetStep * WARP_SIZE; i < numBuckets; i++)
            {
                if (countHistogram[i])
                {
                    targetIdx = i;
                    if ((prev + histPtr[i]) >= currentSum)
                    {
                        break;
                    }
                    prev += histPtr[i];
                }
            }

            counter->sum = currentSum - prev;         // how many values still are there to find
            counter->len = countHistogram[targetIdx]; // cur - prev; // number of values in next pass
            typename hipcub::Traits<T>::UnsignedBits bucket = targetIdx;
            int startBit = calcStartBit<T, BitsPerPass>(pass);
            counter->kthValueBits |= bucket << startBit;
        }
        __syncthreads();

        int constexpr numPasses = calcNumPasses<T, BitsPerPass>();
        // reset for next pass
        if (pass != numPasses - 1)
        {
            for (int i = threadIdx.x; i < numBuckets; i += blockDim.x)
            {
                histogram[i] = 0;
                countHistogram[i] = 0;
            }
        }
        if (threadIdx.x == 0)
        {
            counter->previousLen = currentLen;
            // not necessary for the last pass, but put it here anyway
            counter->filterCnt = 0;
        }

        if (pass == numPasses - 1)
        {
            // Used when isDeterministic==true
            // idxBuf1 and idxBuf2 are ping-pong buffers used in previous iterations to store candidates.
            // In the last pass (pass==2 for single-precision and pass==1 for half-precision),
            // we reuse the buffer didn't store the candidates (idxBuf1 for single-precision and idxBuf2 for
            // half-precision) to help find the correct index of the result.
            [[maybe_unused]] IdxT* lastIdxBuf
                = (pass % 2 == 0) ? idxBuf1 + bufLen * batchId : idxBuf2 + bufLen * batchId;
            if constexpr (isFusedFilter)
            {
                lastFilter<T, IdxT, AccT, BitsPerPass, BlockSize, isDeterministic>(outBuf ? outBuf : inBuf,
                    outIdxBuf ? outIdxBuf : inIdxBuf, outBuf ? currentLen : counter->oriLen, counter, outputLogProbs,
                    cumLogProbs, ids, endIds, sequenceLengths, finishedOutput, batchSlot, maxBatchSize, lastIdxBuf,
                    countHistogramSmem);
                __syncthreads();
            }
        }
    }
}

/**
 * Initialize the Counter<T, IdxT, AccT> and the histogram and countHistogram.
 */
template <typename T, typename IdxT, typename AccT, typename HisT, int BitsPerPass, int BlockSize>
__global__ void airTopPInitialize(Counter<T, IdxT, AccT>* counters, int const batchSize, int const len, T const* in,
    IdxT const* inIdx, float const* topPs, hiprandState_t* hiprandState, float const* randomVals, HisT* histograms,
    IdxT* countHistograms, int32_t const* batchSlots)
{
    auto const batchIdx = blockIdx.x;
    auto const batchSlot = batchSlots ? batchSlots[batchIdx] : batchIdx;
    Counter<T, IdxT, AccT>* counter = counters + batchIdx;
    IdxT offset = batchIdx * len;
    IdxT bufOffset = batchIdx * calcBufLen<T>(len);
    if (threadIdx.x == 0)
    {
        counter->in = in + offset;
        counter->inIdx = nullptr;
        if (inIdx)
        {
            counter->inIdx = inIdx + offset;
        }

        counter->len = len;
        counter->oriLen = len;
        counter->previousLen = len;

        float const probThreshold = topPs[batchSlot];
        auto const randomNumber = randomVals ? randomVals[batchSlot] : hiprand_uniform(hiprandState + batchSlot);
        float const randP = randomNumber * probThreshold;
        counter->p = randP;
        counter->sum = 0;

        counter->kthValueBits = 0;
        counter->finishedBlockCnt = 0;
        counter->filterCnt = 0;
    }

    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    HisT* histogram = histograms + batchIdx * numBuckets;
    for (int i = threadIdx.x; i < numBuckets; i += BlockSize)
    {
        histogram[i] = 0;
    }

    IdxT* countHistogram = nullptr;
    if (countHistograms)
    {
        countHistogram = countHistograms + batchIdx * numBuckets;
        for (int i = threadIdx.x; i < numBuckets; i += BlockSize)
        {
            countHistogram[i] = 0;
        }
    }
}

/*
 *  Calculate the number of blocks based on the batchSize and len to avoid tailing effect.
 */
template <typename T>
uint32_t calcAirTopPBlockNum(int batchSize, int len, int smCnt, bool isDeterministic)
{
    int constexpr BitsPerPass = 11;
    int constexpr BlockSize = 512;
    int constexpr VECTORIZED_READ_SIZE = 16;
    static_assert(VECTORIZED_READ_SIZE / sizeof(T) >= 1);
    TLLM_CHECK_WITH_INFO(
        smCnt > 0, "AIR Top-P needs the count of multiprocessor to calculate the proper block dimension settings");

    int activeBlocks;
    if (isDeterministic)
    {
        using HisT = std::conditional_t<std::is_same_v<T, float>, uint64_t, uint32_t>;
        hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &activeBlocks, airTopPSampling<T, IdxT, AccT, HisT, BitsPerPass, BlockSize, false, true>, BlockSize, 0);
    }
    else
    {
        hipOccupancyMaxActiveBlocksPerMultiprocessor(
            &activeBlocks, airTopPSampling<T, IdxT, AccT, float, BitsPerPass, BlockSize, false, false>, BlockSize, 0);
    }
    activeBlocks *= smCnt;

    IdxT bestNumBlocks = 0;
    float bestTailWavePenalty = 1.0f;
    IdxT const maxNumBlocks = ceilDiv<IdxT>(len, VECTORIZED_READ_SIZE / sizeof(T) * BlockSize);
    for (int numWaves = 1;; ++numWaves)
    {
        IdxT numBlocks = std::min(maxNumBlocks, static_cast<IdxT>(std::max(numWaves * activeBlocks / batchSize, 1)));
        IdxT itemsPerThread = ceilDiv<IdxT>(len, numBlocks * BlockSize);
        itemsPerThread = alignTo<IdxT>(itemsPerThread, VECTORIZED_READ_SIZE / sizeof(T));
        numBlocks = ceilDiv<IdxT>(len, itemsPerThread * BlockSize);
        float actualNumWaves = static_cast<float>(numBlocks) * batchSize / activeBlocks;
        float tailWavePenalty = (ceilf(actualNumWaves) - actualNumWaves) / ceilf(actualNumWaves);

        // 0.15 is determined experimentally. It also ensures breaking the loop
        // early, e.g. when numWaves > 7, tailWavePenalty will always <0.15
        if (tailWavePenalty < 0.15)
        {
            bestNumBlocks = numBlocks;
            break;
        }
        else if (tailWavePenalty < bestTailWavePenalty)
        {
            bestNumBlocks = numBlocks;
            bestTailWavePenalty = tailWavePenalty;
        }

        if (numBlocks == maxNumBlocks)
        {
            break;
        }
    }
    return bestNumBlocks;
}

template <typename T, bool isDeterministic = false>
[[nodiscard]] std::vector<size_t> getAirTopPWorkspaceSizes(int32_t batchSize, int32_t vocabSize)
{
    using HisT
        = std::conditional_t<isDeterministic, std::conditional_t<std::is_same_v<T, float>, uint64_t, uint32_t>, float>;
    int constexpr BitsPerPass = 11;
    int constexpr numBuckets = calcNumBuckets<BitsPerPass>();
    IdxT const bufLen = calcBufLen<T>(vocabSize);

    size_t countersSize = sizeof(Counter<T, IdxT, AccT>) * batchSize;
    size_t histogramsSize = sizeof(HisT) * numBuckets * batchSize;
    size_t countHistogramsSize = sizeof(IdxT) * numBuckets * batchSize;
    size_t buf1Size = sizeof(T) * bufLen * batchSize;
    size_t idxBuf1Size = sizeof(IdxT) * bufLen * batchSize;
    size_t buf2Size = sizeof(T) * bufLen * batchSize;
    size_t idxBuf2Size = sizeof(IdxT) * bufLen * batchSize;

    std::vector<size_t> sizes
        = {countersSize, histogramsSize, countHistogramsSize, buf1Size, idxBuf1Size, buf2Size, idxBuf2Size};

    return sizes;
}

template std::vector<size_t> getAirTopPWorkspaceSizes<float, true>(int32_t batchSize, int32_t vocabSize);
template std::vector<size_t> getAirTopPWorkspaceSizes<float, false>(int32_t batchSize, int32_t vocabSize);
template std::vector<size_t> getAirTopPWorkspaceSizes<half, true>(int32_t batchSize, int32_t vocabSize);
template std::vector<size_t> getAirTopPWorkspaceSizes<half, false>(int32_t batchSize, int32_t vocabSize);

template <typename T, bool isDeterministic = false>
void invokeAirTopPSamplingWithDeterministicPara(TopPSamplingKernelParams<T> const& params, hipStream_t stream)
{
    using HisT
        = std::conditional_t<isDeterministic, std::conditional_t<std::is_same_v<T, float>, uint64_t, uint32_t>, float>;

    static_assert(std::is_same_v<T, half> | std::is_same_v<T, float>, "T needs to be either half or float");
    static_assert(std::is_same_v<AccT, float>, "AccT needs to be float");
    TLLM_CHECK_WITH_INFO(((std::is_same_v<T, half>) &&(params.vocabSizePadded < pow(2, 22)) && isDeterministic)
            || ((std::is_same_v<T, float>) &&(params.vocabSizePadded < pow(2, 41)) && isDeterministic)
            || (!isDeterministic),
        "For Deterministic AIR Top-P, the maximum vocab_size we support is pow(2,22) for half-precision and pow(2,41) "
        "for single-precision");

    IdxT const vocabSize = params.vocabSizePadded;
    int constexpr BitsPerPass = 11;

    int constexpr SAMPLING_BLOCK_SIZE = 512;
    int constexpr THREADS_PER_CTA_TOP_P_INIT = 1024;

    Counter<T, IdxT, AccT>* counters = nullptr;
    HisT* histograms = nullptr;
    IdxT* countHistograms = nullptr;
    T* buf1 = nullptr;
    IdxT* idxBuf1 = nullptr;
    T* buf2 = nullptr;
    IdxT* idxBuf2 = nullptr;

    auto const workspaceSizes = getAirTopPWorkspaceSizes<T, isDeterministic>(params.batchSize, vocabSize);
    calcAlignedPointers(params.workspace, workspaceSizes)(
        counters, histograms, countHistograms, buf1, idxBuf1, buf2, idxBuf2);

    airTopPInitialize<T, IdxT, AccT, HisT, BitsPerPass, THREADS_PER_CTA_TOP_P_INIT>
        <<<params.batchSize, THREADS_PER_CTA_TOP_P_INIT, 0, stream>>>(counters, params.batchSize, vocabSize,
            params.probs, nullptr, params.topPs, params.hiprandState, params.randomVals, histograms, countHistograms,
            params.batchSlots);

    dim3 grid(params.blockNum, params.batchSize);
    // Sample with Top P given sorted tokens
    int constexpr numPasses = calcNumPasses<T, BitsPerPass>();
    auto kernel = airTopPSampling<T, IdxT, AccT, HisT, BitsPerPass, SAMPLING_BLOCK_SIZE, false, isDeterministic>;

    for (int pass = 0; pass < numPasses; ++pass)
    {
        if (pass == numPasses - 1)
        {
            kernel = airTopPSampling<T, IdxT, AccT, HisT, BitsPerPass, SAMPLING_BLOCK_SIZE, true, isDeterministic>;
        }

        kernel<<<grid, SAMPLING_BLOCK_SIZE, 0, stream>>>(counters, histograms, countHistograms, params.outputIdsPtrs,
            params.sequenceLength, params.finishedInput, params.finishedOutput, params.cumLogProbs,
            params.outputLogProbs, params.endIds, params.maxBatchSize, params.skipDecode, pass, buf1, idxBuf1, buf2,
            idxBuf2, params.batchSlots);
    }
}

template <typename T>
void invokeBatchAirTopPSampling(TopPSamplingKernelParams<T> const& params, hipStream_t stream)
{
    if (params.isDeterministic)
    {
        invokeAirTopPSamplingWithDeterministicPara<T, true>(params, stream);
    }
    else
    {
        invokeAirTopPSamplingWithDeterministicPara<T, false>(params, stream);
    }
}

template void invokeBatchAirTopPSampling(TopPSamplingKernelParams<float> const& params, hipStream_t stream);

template void invokeBatchAirTopPSampling(TopPSamplingKernelParams<half> const& params, hipStream_t stream);

template <typename T>
size_t getAirTopPWorkspaceSize(int32_t batchSize, int32_t vocabSizePadded, bool isDeterministic)
{
    std::vector<size_t> workspaceSizes;
    if (isDeterministic == true)
    {
        workspaceSizes = getAirTopPWorkspaceSizes<T, true>(batchSize, vocabSizePadded);
    }
    else
    {
        workspaceSizes = getAirTopPWorkspaceSizes<T, false>(batchSize, vocabSizePadded);
    }
    return calcAlignedSize(workspaceSizes, 256);
}

template size_t getAirTopPWorkspaceSize<float>(int32_t batchSize, int32_t vocabSizePadded, bool isDeterministic);
template size_t getAirTopPWorkspaceSize<half>(int32_t batchSize, int32_t vocabSizePadded, bool isDeterministic);

template uint32_t calcAirTopPBlockNum<float>(int batchSize, int len, int smCnt, bool isDeterministic);
template uint32_t calcAirTopPBlockNum<half>(int batchSize, int len, int smCnt, bool isDeterministic);
} // namespace kernels
} // namespace tensorrt_llm
