#include "hip/hip_runtime.h"
/*
 * MIT License
 *
 * Copyright (c) 2025 DeepSeek
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * reference: https://github.com/deepseek-ai/FlashMLA
 */

#include "flash_fwd_mla_kernel.h"

[[maybe_unused]] static constexpr int MaxBatchSize = 4096;

#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
__launch_bounds__(256, 1, 1)
#endif // __CUDA_ARCH__ >= 900
    __global__ void get_mla_metadata_kernel(__grid_constant__ const Mla_metadata_params params)
{
    int* seqlens_k_ptr = params.seqlens_k_ptr;
    int* tile_scheduler_metadata_ptr = params.tile_scheduler_metadata_ptr;
    int* num_splits_ptr = params.num_splits_ptr;
    int batch_size = params.batch_size;
    int block_size_n = params.block_size_n;
    int fixed_overhead_num_blocks = params.fixed_overhead_num_blocks;
    int num_sm_parts = params.num_sm_parts;

    __shared__ int num_blocks_shared[MaxBatchSize];
    __shared__ int num_splits_shared[MaxBatchSize];

    int total_num_blocks = 0;
    for (int i = threadIdx.x; i < batch_size; i += 32)
    {
        int num_blocks = cutlass::ceil_div(seqlens_k_ptr[i], block_size_n);
        total_num_blocks += num_blocks + fixed_overhead_num_blocks;
        num_blocks_shared[i] = num_blocks;
    }
    for (int offset = 16; offset >= 1; offset /= 2)
    {
        total_num_blocks += __shfl_xor_sync(uint32_t(-1), total_num_blocks, offset);
    }
    __syncwarp();

    if (threadIdx.x == 0)
    {
        int payload = cutlass::ceil_div(total_num_blocks, num_sm_parts) + fixed_overhead_num_blocks;

        int now_idx = 0, now_block = 0, now_n_split_idx = 0, cum_num_splits = 0;
        num_splits_shared[0] = 0;
        for (int i = 0; i < num_sm_parts; ++i)
        {
            int tile_scheduler_metadata0[4], tile_scheduler_metadata1;
            tile_scheduler_metadata0[0] = now_idx;
            tile_scheduler_metadata0[1] = now_block * block_size_n;
            tile_scheduler_metadata1 = now_n_split_idx;
            int remain_payload = payload;
            while (now_idx < batch_size)
            {
                int num_blocks = num_blocks_shared[now_idx];
                int now_remain_blocks = num_blocks - now_block;
                if (remain_payload >= now_remain_blocks + fixed_overhead_num_blocks)
                {
                    cum_num_splits += now_n_split_idx + 1;
                    num_splits_shared[now_idx + 1] = cum_num_splits;
                    remain_payload -= now_remain_blocks + fixed_overhead_num_blocks;
                    ++now_idx;
                    now_block = 0;
                    now_n_split_idx = 0;
                }
                else
                {
                    if (remain_payload - fixed_overhead_num_blocks > 0)
                    {
                        now_block += remain_payload - fixed_overhead_num_blocks;
                        ++now_n_split_idx;
                        remain_payload = 0;
                    }
                    break;
                }
            }
            tile_scheduler_metadata0[2] = now_block > 0 ? now_idx : now_idx - 1;
            tile_scheduler_metadata0[3] = now_block > 0 ? now_block * block_size_n : seqlens_k_ptr[now_idx - 1];
            *reinterpret_cast<int4*>(tile_scheduler_metadata_ptr + i * TileSchedulerMetaDataSize)
                = *reinterpret_cast<int4*>(tile_scheduler_metadata0);
            tile_scheduler_metadata_ptr[i * TileSchedulerMetaDataSize + 4] = tile_scheduler_metadata1;
        }
        FLASH_DEVICE_ASSERT(now_idx == batch_size && now_block == 0 && now_n_split_idx == 0);
    }
    __syncwarp();

    for (int i = threadIdx.x; i <= batch_size; i += 32)
    {
        num_splits_ptr[i] = num_splits_shared[i];
    }
}

void get_mla_metadata_func(Mla_metadata_params& params, hipStream_t stream)
{
    FLASH_ASSERT(params.batch_size < MaxBatchSize);
    get_mla_metadata_kernel<<<1, 32, 0, stream>>>(params);
    CHECK_CUDA_KERNEL_LAUNCH();
}
