/*
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "flash_fwd_mla_kernel.h"

template void run_mha_fwd_splitkv_mla<cutlass::float_e4m3_t, cutlass::bfloat16_t, 576>(
    Flash_fwd_mla_params& params, hipStream_t stream);
