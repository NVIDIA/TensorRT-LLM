#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "fusedQKNormRopeKernel.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/mathUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include <cmath>
#include <hip/hip_bf16.h>
#include <hip/hip_fp16.h>
#include <hip/hip_fp8.h>
#include <hip/hip_runtime.h>

namespace tensorrt_llm::common
{
// Specialization for packed_as used in this kernel.
template <>
struct packed_as<uint, 1>
{
    using type = uint;
};

template <>
struct packed_as<uint, 2>
{
    using type = uint2;
};

template <>
struct packed_as<uint, 4>
{
    using type = uint4;
};
} // namespace tensorrt_llm::common

namespace tensorrt_llm::kernels
{

////////////////////////////////////////////////////////////////////////////////////////////////////

// Perform per-head QK Norm and RoPE in a single kernel.
// head_dim: the dimension of each head
// interleave: interleave=!is_neox.
template <int head_dim, bool interleave>
__global__ void fusedQKNormRopeKernel(
    __hip_bfloat16* qkv,            // Combined QKV tensor [num_tokens, (num_heads_q+num_heads_k+num_heads_v)*head_dim]
    int const num_heads_q,         // Number of query heads
    int const num_heads_k,         // Number of key heads
    int const num_heads_v,         // Number of value heads
    float const eps,               // Epsilon for RMS normalization
    __hip_bfloat16 const* q_weight, // RMSNorm weights for query
    __hip_bfloat16 const* k_weight, // RMSNorm weights for key
    float const base,              // Base for RoPE computation
    int const* position_ids,       // Position IDs for RoPE
    int const num_tokens           // Number of tokens
)
{
    int const warpsPerBlock = blockDim.x / 32;
    int const warpId = threadIdx.x / 32;
    int const laneId = threadIdx.x % 32;

    // Calculate global warp index to determine which head/token this warp processes
    int const globalWarpIdx = blockIdx.x * warpsPerBlock + warpId;

    // Total number of attention heads (Q and K)
    int const total_qk_heads = num_heads_q + num_heads_k;

    // Determine which token and head type (Q or K) this warp processes
    int const tokenIdx = globalWarpIdx / total_qk_heads;
    int const localHeadIdx = globalWarpIdx % total_qk_heads;

    // Skip if this warp is assigned beyond the number of tokens
    if (tokenIdx >= num_tokens)
        return;

    bool const isQ = localHeadIdx < num_heads_q;
    int const headIdx = isQ ? localHeadIdx : localHeadIdx - num_heads_q;

    int const num_heads = num_heads_q + num_heads_k + num_heads_v;

    static_assert(head_dim % (32 * 2) == 0,
        "head_dim must be divisible by 64 (each warp processes one head, and each thread gets even number of "
        "elements)");
    constexpr int numElemsPerThread = head_dim / 32;
    float elements[numElemsPerThread];
    constexpr int elemSizeBytes = numElemsPerThread * sizeof(__hip_bfloat16);
    static_assert(elemSizeBytes % 4 == 0, "numSizeBytes must be a multiple of 4");
    constexpr int vecSize = elemSizeBytes / 4; // Use packed_as<uint, vecSize> to perform loading/saving.
    using vec_T = typename tensorrt_llm::common::packed_as<uint, vecSize>::type;

    int offsetWarp; // Offset for the warp
    if (isQ)
    {
        // Q segment: token offset + head offset within Q segment
        offsetWarp = tokenIdx * num_heads * head_dim + headIdx * head_dim;
    }
    else
    {
        // K segment: token offset + entire Q segment + head offset within K segment
        offsetWarp = tokenIdx * num_heads * head_dim + num_heads_q * head_dim + headIdx * head_dim;
    }
    int offsetThread = offsetWarp + laneId * numElemsPerThread;

    // Sum of squares for RMSNorm
    float sumOfSquares = 0.0f;

    // Load.
    {
        vec_T vec = *reinterpret_cast<vec_T const*>(&qkv[offsetThread]);
        for (int i = 0; i < vecSize; i++)
        {
            float2 vals = __bfloat1622float2(*reinterpret_cast<__hip_bfloat162*>(reinterpret_cast<uint*>(&vec) + i));
            sumOfSquares += vals.x * vals.x;
            sumOfSquares += vals.y * vals.y;

            elements[2 * i] = vals.x;
            elements[2 * i + 1] = vals.y;
        }
    }

    // Reduce sum across warp using the utility function
    sumOfSquares = tensorrt_llm::common::warpReduceSum(sumOfSquares);

    // Compute RMS normalization factor
    float rms_rcp = rsqrtf(sumOfSquares / static_cast<float>(head_dim) + eps);

    // Normalize elements
    for (int i = 0; i < numElemsPerThread; i++)
    {
        int dim = laneId * numElemsPerThread + i;
        float weight = isQ ? __bfloat162float(q_weight[dim]) : __bfloat162float(k_weight[dim]);
        elements[i] *= rms_rcp * weight;
    }

    // Apply RoPE to normalized elements
    float elements2[numElemsPerThread]; // Additional buffer required for RoPE.
    float cos_vals[numElemsPerThread];
    float sin_vals[numElemsPerThread];

    float pos_id = static_cast<float>(position_ids[tokenIdx]);

    // TODO: cos sin calculation could be halved.
    if constexpr (interleave)
    {
        // Perform interleaving. Fill cos_vals and sin_vals.
        for (int i = 0; i < numElemsPerThread; i++)
        {
            if (i % 2 == 0)
            {
                elements2[i] = -elements[i + 1];
            }
            else
            {
                elements2[i] = elements[i - 1];
            }

            int dim_idx = laneId * numElemsPerThread + i;
            int half_dim = dim_idx / 2;
            float freq = powf(base, -2.0f * half_dim / static_cast<float>(head_dim));
            float theta = pos_id * freq;
            __sincosf(theta, &sin_vals[i], &cos_vals[i]);
        }
    }
    else
    {
        // Before data exchange with in warp, we need to sync.
        __syncwarp();
        // Get the data from the other half of the warp. Fill cos_vals and sin_vals.
        for (int i = 0; i < numElemsPerThread; i++)
        {
            elements2[i] = __shfl_xor_sync(0xffffffff, elements[i], 16);
            if (laneId < 16)
            {
                elements2[i] = -elements2[i];
            }

            int dim_idx = laneId * numElemsPerThread + i;
            dim_idx = (dim_idx * 2) % head_dim;
            int half_dim = dim_idx / 2;
            float freq = powf(base, -2.0f * half_dim / static_cast<float>(head_dim));
            float theta = pos_id * freq;
            __sincosf(theta, &sin_vals[i], &cos_vals[i]);
        }
        // __shfl_xor_sync does not provide memfence. Need to sync again.
        __syncwarp();
    }

    for (int i = 0; i < numElemsPerThread; i++)
    {
        elements[i] = elements[i] * cos_vals[i] + elements2[i] * sin_vals[i];
    }

    // Store.
    {
        vec_T vec;
        for (int i = 0; i < vecSize; i++)
        {
            __hip_bfloat162 vals = __float22bfloat162_rn(make_float2(elements[2 * i], elements[2 * i + 1]));
            reinterpret_cast<__hip_bfloat162&>(*(reinterpret_cast<uint*>(&vec) + i)) = vals;
        }
        vec_T* outputPtr = reinterpret_cast<vec_T*>(&qkv[offsetThread]);
        *outputPtr = vec;
    }
}

// Borrowed from
// https://github.com/flashinfer-ai/flashinfer/blob/8125d079a43e9a0ba463a4ed1b639cefd084cec9/include/flashinfer/pos_enc.cuh#L568
#define DISPATCH_INTERLEAVE(interleave, INTERLEAVE, ...)                                                               \
    if (interleave)                                                                                                    \
    {                                                                                                                  \
        const bool INTERLEAVE = true;                                                                                  \
        __VA_ARGS__                                                                                                    \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        const bool INTERLEAVE = false;                                                                                 \
        __VA_ARGS__                                                                                                    \
    }

void launchFusedQKNormRope(void* qkv, int const num_tokens, int const num_heads_q, int const num_heads_k,
    int const num_heads_v, int const head_dim, float const eps, void const* q_weight, void const* k_weight,
    float const base, bool const interleave, int const* position_ids, hipStream_t stream)
{
    constexpr int blockSize = 256;

    int const warpsPerBlock = blockSize / 32;
    int const totalQKHeads = num_heads_q + num_heads_k;
    int const totalWarps = num_tokens * totalQKHeads;

    int const gridSize = common::divUp(totalWarps, warpsPerBlock);
    dim3 gridDim(gridSize);
    dim3 blockDim(blockSize);

    // Head dimensions should be a multiple of 64
    // Add more cases as needed
    switch (head_dim)
    {
    case 64:
        DISPATCH_INTERLEAVE(interleave, INTERLEAVE, {
            fusedQKNormRopeKernel<64, INTERLEAVE>
                <<<gridDim, blockDim, 0, stream>>>(reinterpret_cast<__hip_bfloat16*>(qkv), num_heads_q, num_heads_k,
                    num_heads_v, eps, reinterpret_cast<__hip_bfloat16 const*>(q_weight),
                    reinterpret_cast<__hip_bfloat16 const*>(k_weight), base, position_ids, num_tokens);
        });
        break;
    case 128:
        DISPATCH_INTERLEAVE(interleave, INTERLEAVE, {
            fusedQKNormRopeKernel<128, INTERLEAVE>
                <<<gridDim, blockDim, 0, stream>>>(reinterpret_cast<__hip_bfloat16*>(qkv), num_heads_q, num_heads_k,
                    num_heads_v, eps, reinterpret_cast<__hip_bfloat16 const*>(q_weight),
                    reinterpret_cast<__hip_bfloat16 const*>(k_weight), base, position_ids, num_tokens);
        });
        break;
    default: TLLM_THROW("Unsupported head dimension for fusedQKNormRope: %d", head_dim);
    }
}
} // namespace tensorrt_llm::kernels
