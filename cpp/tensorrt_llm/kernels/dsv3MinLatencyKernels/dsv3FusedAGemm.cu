#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 * Copyright (c) 2021, NAVER Corp.  Authored by CLOVA.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdio>
#include <tuple>

#include "hip/hip_runtime.h"
#include "hip/hip_bf16.h"
#include "hip/hip_runtime.h"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/kernels/dsv3MinLatencyKernels/dsv3FusedAGemm.h"

using namespace tensorrt_llm::common;
using bf16_t = __hip_bfloat16;

using namespace tensorrt_llm::common;

namespace tensorrt_llm::kernels::dsv3MinLatencyKernels
{

__device__ void hmma_16_8_16_f32acc_bf16ab(
    float (&d_reg)[4], const bf16_t (&a_reg)[8], const bf16_t (&b_reg)[4], float const (&c_reg)[4])
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    uint32_t a0 = *reinterpret_cast<uint32_t const*>(a_reg + 0);
    uint32_t a1 = *reinterpret_cast<uint32_t const*>(a_reg + 2);
    uint32_t a2 = *reinterpret_cast<uint32_t const*>(a_reg + 4);
    uint32_t a3 = *reinterpret_cast<uint32_t const*>(a_reg + 6);
    uint32_t b0 = *reinterpret_cast<uint32_t const*>(b_reg + 0);
    uint32_t b1 = *reinterpret_cast<uint32_t const*>(b_reg + 2);
    asm volatile(
        "mma.sync.aligned.m16n8k16.row.col.f32.bf16.bf16.f32 "
        "{%0,  %1,  %2,  %3},"
        "{%4,  %5,  %6,  %7},"
        "{%8,  %9},"
        "{%10, %11, %12, %13};\n"
        : "=f"(d_reg[0]), "=f"(d_reg[1]), "=f"(d_reg[2]), "=f"(d_reg[3])
        : "r"(a0), "r"(a1), "r"(a2), "r"(a3), "r"(b0), "r"(b1), "f"(d_reg[0]), "f"(d_reg[1]), "f"(d_reg[2]),
        "f"(d_reg[3]));
#endif
}

extern "C"
{
    __device__ uint32_t __nvvm_get_smem_pointer(void*);
}

__device__ void ldgsts_128(void const* gPtr, void* sPtr, uint32_t pred)
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    if (pred)
    {
        uint32_t smemPtrAsUint32 = __nvvm_get_smem_pointer(sPtr);
        asm volatile("cp.async.cg.shared.global.L2::128B [%0], [%1], %2;\n" ::"r"(smemPtrAsUint32), "l"(gPtr), "n"(16));
    }
#endif
}

__device__ void ldsm_x4(void* smem_ptr, uint32_t* reg_ptr)
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    asm volatile("ldmatrix.sync.aligned.x4.m8n8.shared.b16 {%0, %1, %2, %3}, [%4];\n"
                 : "=r"(reg_ptr[0]), "=r"(reg_ptr[1]), "=r"(reg_ptr[2]), "=r"(reg_ptr[3])
                 : "r"(__nvvm_get_smem_pointer(smem_ptr)));
#endif
}

template <class Type>
__device__ int apply_swizzle_343_on_elem_row_col(int row_idx_, int col_idx_)
{
    uint32_t row_idx = *reinterpret_cast<uint32_t*>(&row_idx_);
    uint32_t col_idx = *reinterpret_cast<uint32_t*>(&col_idx_);
    row_idx = row_idx % 8;
    row_idx = row_idx * (16 / sizeof(Type));
    col_idx = col_idx ^ row_idx;
    return *reinterpret_cast<int*>(&col_idx);
}

__device__ void initialize_barrier(uint64_t* smem_barrier, // 64 bits user-manged barrier in smem
    int thread_count = 1)                                  // Thread count expected to arrive/wait on this barrier
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    uint32_t smem_int_ptr = __nvvm_get_smem_pointer(smem_barrier);
    asm volatile("mbarrier.init.shared::cta.b64 [%0], %1;\n" ::"r"(smem_int_ptr), "r"(thread_count));
#endif
}

// Barrier wait
__device__ void wait_barrier(uint64_t* smem_barrier, // 64 bits user-manged barrier in smem
    int phase_bit)                                   // Current phase bit the barrier waiting to flip
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    uint32_t smem_int_ptr = __nvvm_get_smem_pointer(smem_barrier);
    asm volatile(
        "{\n"
        ".reg .pred                P1;\n"
        "LAB_WAIT:\n"
        "mbarrier.try_wait.parity.shared::cta.b64 P1, [%0], %1;\n"
        "@P1                       bra DONE;\n"
        "bra                   LAB_WAIT;\n"
        "DONE:\n"
        "}\n" ::"r"(smem_int_ptr),
        "r"(phase_bit));
#endif
}

__device__ bool try_wait_barrier(uint64_t* smem_ptr, int phase_bit)
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    uint32_t wait_complete;
    uint32_t smem_int_ptr = __nvvm_get_smem_pointer(smem_ptr);
    asm volatile(
        "{\n\t"
        ".reg .pred P1; \n\t"
        "mbarrier.try_wait.parity.shared::cta.b64 P1, [%1], %2; \n\t"
        "selp.b32 %0, 1, 0, P1; \n\t"
        "}"
        : "=r"(wait_complete)
        : "r"(smem_int_ptr), "r"(phase_bit));
    return static_cast<bool>(wait_complete);
#endif
}

// Barrier arrive
__device__ void arrive_barrier(uint64_t* smem_barrier) // 64 bits user-manged barrier in smem
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    uint32_t smem_int_ptr = __nvvm_get_smem_pointer(smem_barrier);
    asm volatile(
        "{\n"
        ".reg .b64 state; \n"
        "mbarrier.arrive.shared::cta.b64   state, [%0];\n"
        "}\n" ::"r"(smem_int_ptr));
#endif
}

__device__ void ldgsts_arrive(uint64_t* smem_barrier)
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    uint32_t smem_int_ptr = __nvvm_get_smem_pointer(smem_barrier);
    asm volatile("cp.async.mbarrier.arrive.noinc.shared.b64 [%0];" : : "r"(smem_int_ptr));
#endif
}

template <int gemm_k, int tile_m, int tile_k, int stage_cnt>
struct GmemLoaderA
{
    static constexpr int elem_bytes = 2;
    static constexpr int vec_bytes = 16;
    static constexpr int vec_elems = vec_bytes / elem_bytes;
    static constexpr int thread_cnt = 64;
    static_assert((tile_m * tile_k) % (vec_elems * thread_cnt) == 0);
    static constexpr int a_inst_cnt_per_iter = (tile_m * tile_k) / (vec_elems * thread_cnt);
    static_assert(gemm_k % tile_k == 0);
    static constexpr int k_iter_cnt = gemm_k / tile_k;

    // Extra params to keep the order of k reduction...
    static constexpr int mma_warp_cnt = 4;
    static constexpr int per_mma_warp_k = tile_k / mma_warp_cnt;
    static constexpr int k_each_chunk = gemm_k / mma_warp_cnt;

private:
    __device__ int k_project(int tile_k_idx)
    {
        return (tile_k_idx / per_mma_warp_k * k_each_chunk) + (tile_k_idx % per_mma_warp_k);
    }

public:
    __device__ GmemLoaderA(bf16_t const* gmem_a_local_, bf16_t* smem_a_, uint64_t* smem_barrier_)
        : gmem_a(gmem_a_local_)
        , smem_a(smem_a_)
        , smem_barrier(smem_barrier_)
        , local_tid(threadIdx.x % thread_cnt)
    {
    }

    __device__ void prepare()
    {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
// swizzle, that's what we want.
#pragma unroll
        for (int i = 0; i < a_inst_cnt_per_iter; i++)
        {
            int linear_idx = local_tid * vec_elems + i * thread_cnt * vec_elems;
            int m_idx = linear_idx / tile_k;
            int k_idx = linear_idx % tile_k;
            k_idx = apply_swizzle_343_on_elem_row_col<bf16_t>(m_idx, k_idx);
            a_smem_offsets[i] = m_idx * tile_k + k_idx;
        }
#endif
    }

    __device__ void issue_mainloop()
    {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
#pragma unroll 1
        for (int loop_idx = 0; loop_idx < k_iter_cnt; loop_idx++)
        {
            if (need_wait)
            {
                wait_barrier(smem_barrier + 1 + stage_idx * 2, phase_bit);
            }
            int next_stage_idx = stage_idx + 1;
            int next_phase_bit = next_stage_idx == stage_cnt ? phase_bit ^ 1 : phase_bit;
            next_stage_idx = next_stage_idx == stage_cnt ? 0 : next_stage_idx;
            if (loop_idx != k_iter_cnt - 1)
            {
                need_wait = !try_wait_barrier(smem_barrier + 1 + next_stage_idx * 2, next_phase_bit);
            }

#pragma unroll
            for (int i = 0; i < a_inst_cnt_per_iter; i++)
            {
                int smem_offset = a_smem_offsets[i];
                bf16_t* smem_ptr_this_iter = smem_a + stage_idx * tile_m * tile_k + smem_offset;
                int linear_idx = local_tid * vec_elems + i * thread_cnt * vec_elems;
                int m_idx = linear_idx / tile_k;
                int k_idx = linear_idx % tile_k;
                int gmem_offset = m_idx * gemm_k + k_project(k_idx);
                bf16_t const* gmem_ptr_this_iter = gmem_a + gmem_offset;
                ldgsts_128(gmem_ptr_this_iter, smem_ptr_this_iter, true);
            }
            ldgsts_arrive(smem_barrier + stage_idx * 2);

            stage_idx = next_stage_idx;
            phase_bit = next_phase_bit;
            gmem_a += per_mma_warp_k;
        }
#endif
    }

    bf16_t const* gmem_a;
    bf16_t* smem_a;
    uint64_t* smem_barrier;
    int local_tid;
    int stage_idx = 0;
    int phase_bit = 1;
    bool need_wait = true;

    // per smem_stage, store with swizzle information
    int a_smem_offsets[a_inst_cnt_per_iter];
};

template <int gemm_k, int tile_n, int tile_k, int stage_cnt>
struct GmemLoaderB
{
    static constexpr int elem_bytes = 2;
    static constexpr int vec_bytes = 16;
    static constexpr int vec_elems = vec_bytes / elem_bytes;
    static constexpr int thread_cnt = 64;
    static_assert((tile_n * tile_k) % (vec_elems * thread_cnt) == 0);
    static constexpr int b_inst_cnt_per_iter = (tile_n * tile_k) / (vec_elems * thread_cnt);
    static_assert(gemm_k % tile_k == 0);
    static constexpr int k_iter_cnt = gemm_k / tile_k;

    // Extra params to keep the order of k reduction...
    static constexpr int mma_warp_cnt = 4;
    static constexpr int per_mma_warp_k = tile_k / mma_warp_cnt;
    static constexpr int k_each_chunk = gemm_k / mma_warp_cnt;

private:
    __device__ int k_project(int tile_k_idx)
    {
        return (tile_k_idx / per_mma_warp_k * k_each_chunk) + (tile_k_idx % per_mma_warp_k);
    }

public:
    __device__ GmemLoaderB(bf16_t const* gmem_b_local_, bf16_t* smem_b_, uint64_t* smem_barrier_, int gemm_n_)
        : gmem_b(gmem_b_local_)
        , smem_b(smem_b_)
        , smem_barrier(smem_barrier_)
        , gemm_n(gemm_n_)
        , local_tid(threadIdx.x % thread_cnt)
    {
    }

    __device__ void prepare()
    {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
// swizzle, that's what we want.
#pragma unroll
        for (int i = 0; i < b_inst_cnt_per_iter; i++)
        {
            int linear_idx = local_tid * vec_elems + i * thread_cnt * vec_elems;
            int n_idx = linear_idx / tile_k;
            int k_idx = linear_idx % tile_k;
            k_idx = apply_swizzle_343_on_elem_row_col<bf16_t>(n_idx, k_idx);
            b_smem_offsets[i] = n_idx * tile_k + k_idx;
            preds[i] = n_idx < gemm_n;
        }
#endif
    }

    __device__ void issue_mainloop()
    {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
        asm volatile("griddepcontrol.wait;");
#pragma unroll 1
        for (int loop_idx = 0; loop_idx < k_iter_cnt; loop_idx++)
        {
            if (need_wait)
            {
                wait_barrier(smem_barrier + 1 + stage_idx * 2, phase_bit);
            }
            int next_stage_idx = stage_idx + 1;
            int next_phase_bit = next_stage_idx == stage_cnt ? phase_bit ^ 1 : phase_bit;
            next_stage_idx = next_stage_idx == stage_cnt ? 0 : next_stage_idx;
            if (loop_idx != k_iter_cnt - 1)
            {
                need_wait = !try_wait_barrier(smem_barrier + 1 + next_stage_idx * 2, next_phase_bit);
            }
#pragma unroll
            for (int i = 0; i < b_inst_cnt_per_iter; i++)
            {
                int smem_offset = b_smem_offsets[i];
                bf16_t* smem_ptr_this_iter = smem_b + stage_idx * tile_n * tile_k + smem_offset;
                int linear_idx = local_tid * vec_elems + i * thread_cnt * vec_elems;
                int n_idx = linear_idx / tile_k;
                int k_idx = linear_idx % tile_k;
                int gmem_offset = n_idx * gemm_k + k_project(k_idx);
                bf16_t const* gmem_ptr_this_iter = gmem_b + gmem_offset;
                ldgsts_128(gmem_ptr_this_iter, smem_ptr_this_iter, preds[i]);
            }
            ldgsts_arrive(smem_barrier + stage_idx * 2);

            stage_idx = next_stage_idx;
            phase_bit = next_phase_bit;
            gmem_b += per_mma_warp_k;
        }
#endif
    }

    bf16_t const* gmem_b;
    bf16_t* smem_b;
    uint64_t* smem_barrier;
    int gemm_n;
    int local_tid;
    int stage_idx = 0;
    int phase_bit = 1;
    bool need_wait = true;

    // per smem_stage, store with swizzle information
    int b_smem_offsets[b_inst_cnt_per_iter];
    uint32_t preds[b_inst_cnt_per_iter];
};

template <int gemm_m, int gemm_k, int tile_m, int tile_n, int tile_k, int stage_cnt>
struct MmaComputer
{
    static constexpr int elem_bytes = 2;
    static constexpr int thread_cnt = 128;
    static_assert(gemm_k % tile_k == 0);
    static_assert(tile_k % (thread_cnt / 32) == 0);
    static constexpr int per_warp_tile_k = tile_k / (thread_cnt / 32);
    static constexpr int k_iter_cnt = gemm_k / tile_k;
    static constexpr int k_phase_cnt = per_warp_tile_k / 16;
    static constexpr int m_iter_cnt = (tile_m + 15) / 16;
    static constexpr int n_iter_cnt = (tile_n + 7) / 8; // Possible to have non-1 n_iter_cnt for ab_swap m16 case.
    static_assert(m_iter_cnt == 1);
    static_assert(n_iter_cnt == 1 || n_iter_cnt == 2);

    __device__ MmaComputer(
        bf16_t* gmem_c_local_, bf16_t* smem_a_, bf16_t* smem_b_, uint64_t* smem_barrier_, int warp_idx_, int gemm_n_)
        : gmem_c(gmem_c_local_)
        , smem_a(smem_a_)
        , smem_b(smem_b_)
        , smem_barrier(smem_barrier_)
        , warp_idx(warp_idx_ - (thread_cnt / 32))
        , gemm_n(gemm_n_)
    {
    }

private:
    __device__ constexpr int internal_b_atom_func(int tid)
    {
        if constexpr (tile_n < 8)
        {
            return (tid % tile_n) + ((tid % 8) / tile_n * 0) + tid / 8 * 8 * tile_n;
        }
        else
        {
            return (tid % 8) + ((tid % 32) / 8 * (tile_n * 8));
        }
    }

public:
    __device__ void prepare()
    {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
#pragma unroll
        for (int i = 0; i < k_phase_cnt; i++)
        {
            int linear_idx = (lane_idx % 16) + (lane_idx / 16) * 128 + i * 256;
            int m_idx = linear_idx % tile_m;
            int k_idx = linear_idx / tile_m + warp_k_offset_in_tile_k;
            k_idx = apply_swizzle_343_on_elem_row_col<bf16_t>(m_idx, k_idx);
            a_smem_offsets[0][i] = m_idx * tile_k + k_idx;
        }
#pragma unroll
        for (int n_iter_idx = 0; n_iter_idx < n_iter_cnt; n_iter_idx++)
        {
#pragma unroll
            for (int i = 0; i < k_phase_cnt; i += 2)
            { // Special i+=2 for B.
                int linear_idx = internal_b_atom_func(lane_idx) + i * tile_n * 16 + n_iter_idx * 8;
                int n_idx = linear_idx % tile_n;
                int k_idx = linear_idx / tile_n + warp_k_offset_in_tile_k;
                k_idx = apply_swizzle_343_on_elem_row_col<bf16_t>(n_idx, k_idx);
                b_smem_offsets[n_iter_idx][i] = n_idx * tile_k + k_idx;
            }
        }
#endif
    }

    __device__ void issue_mainloop()
    {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
#pragma unroll 1
        for (int loop_idx = 0; loop_idx < k_iter_cnt; loop_idx++)
        {
            wait_barrier(smem_barrier + 0 + stage_idx * 2, phase_bit);

#pragma unroll
            for (int i = 0; i < k_phase_cnt; i++)
            {
                int smem_offset = a_smem_offsets[0][i];
                bf16_t* smem_ptr_this_iter = smem_a + stage_idx * tile_m * tile_k + smem_offset;
                ldsm_x4(smem_ptr_this_iter, reinterpret_cast<uint32_t*>(a_reg[0][i]));
            }

#pragma unroll
            for (int n_iter_idx = 0; n_iter_idx < n_iter_cnt; n_iter_idx++)
            {
#pragma unroll
                for (int i = 0; i < k_phase_cnt; i += 2)
                {
                    int smem_offset = b_smem_offsets[n_iter_idx][i];
                    bf16_t* smem_ptr_this_iter = smem_b + stage_idx * tile_n * tile_k + smem_offset;
                    ldsm_x4(smem_ptr_this_iter, reinterpret_cast<uint32_t*>(b_reg[n_iter_idx][i]));
                }
            }

#pragma unroll
            for (int k_iter_idx = 0; k_iter_idx < k_phase_cnt; k_iter_idx++)
            {
#pragma unroll
                for (int n_iter_idx = 0; n_iter_idx < n_iter_cnt; n_iter_idx++)
                {
                    hmma_16_8_16_f32acc_bf16ab(acc_reg[0][n_iter_idx], a_reg[0][k_iter_idx],
                        b_reg[n_iter_idx][k_iter_idx], acc_reg[0][n_iter_idx]);
                }
            }
            ::arrive_barrier(smem_barrier + 1 + stage_idx * 2);
            stage_idx += 1;
            phase_bit = stage_idx == stage_cnt ? phase_bit ^ 1 : phase_bit;
            stage_idx = stage_idx == stage_cnt ? 0 : stage_idx;
        }
#endif
    }

    __device__ void epi()
    {
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
        asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(thread_cnt));
        // reorganize the acc_reg
        constexpr int thread_m = 2;
        constexpr int thread_n = 2 * n_iter_cnt;
        constexpr int cta_mma_n = n_iter_cnt * 8;
        float acc_reg_reorg[thread_m][thread_n];

        for (int i = 0; i < thread_m; i++)
        {
            for (int j = 0; j < thread_n; j++)
            {
                acc_reg_reorg[i][j] = acc_reg[0][j / 2][(j % 2) + (i * 2)];
            }
        }

        // 4 x cosize(smem_c_layout)
        float* smem_c = reinterpret_cast<float*>(smem_a);
        // coord -> index
        auto smem_c_index_func = [&](int m_idx, int n_idx)
        {
            int group_rows = 32 / cta_mma_n;
            int group_cnt = 2;
            return (m_idx % group_rows * cta_mma_n) + (m_idx / group_rows * (32 + group_cnt)) + n_idx;
        };
        constexpr int cosize_smem_c = ((tile_m * cta_mma_n) / 32) * (32 + 2);

// This should be optimized to STS.64 but can not be STS.128 due to the bank index.
#pragma unroll
        for (int m_idx_thread = 0; m_idx_thread < thread_m; m_idx_thread++)
        {
#pragma unroll
            for (int n_idx_thread = 0; n_idx_thread < thread_n; n_idx_thread++)
            {
                int m_idx = (lane_idx / 4) + m_idx_thread * 8;
                int n_idx = ((lane_idx % 4) * 2) + (n_idx_thread % 2) + (n_idx_thread / 2) * 8;
                smem_c[cosize_smem_c * warp_idx + smem_c_index_func(m_idx, n_idx)]
                    = acc_reg_reorg[m_idx_thread][n_idx_thread];
            }
        }
        asm volatile("bar.sync %0, %1;" : : "r"(1), "r"(thread_cnt));

        if (warp_idx == 0)
        {
            constexpr int final_acc_reg_cnt = (tile_m * tile_n + 31) / 32;
            float acc_final[final_acc_reg_cnt]{};

#pragma unroll
            for (int reg_idx = 0; reg_idx < final_acc_reg_cnt; reg_idx++)
            {
                int linear_idx = reg_idx * 32 + lane_idx;
                int m_idx = linear_idx % tile_m;
                int n_idx = linear_idx / tile_m;
                acc_final[reg_idx] += smem_c[smem_c_index_func(m_idx, n_idx) + 0 * cosize_smem_c]
                    + smem_c[smem_c_index_func(m_idx, n_idx) + 1 * cosize_smem_c]
                    + smem_c[smem_c_index_func(m_idx, n_idx) + 2 * cosize_smem_c]
                    + smem_c[smem_c_index_func(m_idx, n_idx) + 3 * cosize_smem_c];
            }

#pragma unroll
            for (int reg_idx = 0; reg_idx < final_acc_reg_cnt; reg_idx++)
            {
                int linear_idx = reg_idx * 32 + lane_idx;
                int m_idx = linear_idx % tile_m;
                int n_idx = linear_idx / tile_m;
                if (m_idx < tile_m && n_idx < gemm_n)
                {
                    gmem_c[n_idx * gemm_m + m_idx] = acc_final[reg_idx];
                }
            }
        }
#endif
    }

    bf16_t* gmem_c;
    bf16_t* smem_a;
    bf16_t* smem_b;
    uint64_t* smem_barrier;
    int warp_idx;
    int gemm_n;
    int stage_idx = 0;
    int phase_bit = 0;
    int lane_idx = threadIdx.x % 32;
    int warp_k_offset_in_tile_k = warp_idx * per_warp_tile_k;

    int a_smem_offsets[m_iter_cnt][k_phase_cnt];
    int b_smem_offsets[n_iter_cnt][k_phase_cnt];

    bf16_t a_reg[m_iter_cnt][k_phase_cnt][8];
    bf16_t b_reg[n_iter_cnt][k_phase_cnt][4];
    float acc_reg[m_iter_cnt][n_iter_cnt][4]{};
};

// AB swapped, kernel is k-major, k-major, m-major
template <int batch_size, int gemm_m, int gemm_k, int tile_m, int tile_n, int tile_k, int stage_cnt>
__global__ __launch_bounds__(256, 1) void fused_a_gemm_kernel(
    bf16_t* output, bf16_t const* mat_a, bf16_t const* mat_b, int gemm_n)
{
#if defined(__CUDA_ARCH__) && __CUDA_ARCH__ >= 900
    constexpr int load_thread_cnt = 128;
    constexpr int compute_thread_cnt = 128;
    constexpr int thread_cnt = load_thread_cnt + compute_thread_cnt;
    (void) thread_cnt;
    static_assert(gemm_m % 16 == 0);
    static_assert(gemm_k % tile_k == 0);
    static_assert(gemm_m % tile_m == 0);
    static_assert(tile_k == 128 || tile_k == 256 || tile_k == 512
        || tile_k == 1024); // tile_k must be larger than 64 since 4 warp splitK.
    static_assert(tile_m == 16);
    constexpr int g2s_vec_bytes = 16;
    constexpr int a_elem_bytes = 2;
    constexpr int b_elem_bytes = 2;
    // constexpr int c_elem_bytes = 2;
    static_assert((tile_m * a_elem_bytes + tile_n * b_elem_bytes) * tile_k * stage_cnt <= 225 * 1024);
    static_assert((tile_m * tile_k * a_elem_bytes) % (load_thread_cnt * g2s_vec_bytes) == 0);
    static_assert((tile_n * tile_k * b_elem_bytes) % (load_thread_cnt * g2s_vec_bytes) == 0);

    extern __shared__ char smem[];
    uint64_t* smem_barrier = reinterpret_cast<uint64_t*>(smem); // producer,consumer; producer,consumer; ...
    bf16_t* smem_a = reinterpret_cast<bf16_t*>(smem + (stage_cnt * 8 * 2 + 1024) / 1024 * 1024);
    bf16_t* smem_b = smem_a + tile_m * tile_k * stage_cnt;

    int cta_m_idx = tile_m * blockIdx.x;
    int cta_n_idx = tile_n * blockIdx.y;
    bf16_t const* gmem_a_local = mat_a + cta_m_idx * gemm_k;
    bf16_t const* gmem_b_local = mat_b + cta_n_idx * gemm_k;
    bf16_t* gmem_c_local = output + cta_n_idx * gemm_m + cta_m_idx;

    int warp_idx = __shfl_sync(0xffffffff, threadIdx.x / 32, 0);

    if (warp_idx == 4)
    {
        for (int i = 0; i < stage_cnt; i++)
        {
            initialize_barrier(smem_barrier + i * 2 + 0, load_thread_cnt);    // producer
            initialize_barrier(smem_barrier + i * 2 + 1, compute_thread_cnt); // consumer
        }
    }
    __syncthreads();

    if (warp_idx < 2)
    {
        GmemLoaderA<gemm_k, tile_m, tile_k, stage_cnt> a_loader(gmem_a_local, smem_a, smem_barrier);
        a_loader.prepare();
        a_loader.issue_mainloop();
    }
    else if (warp_idx < 4)
    {
        GmemLoaderB<gemm_k, tile_n, tile_k, stage_cnt> b_loader(gmem_b_local, smem_b, smem_barrier, gemm_n);
        b_loader.prepare();
        b_loader.issue_mainloop();
    }
    else
    {
        MmaComputer<gemm_m, gemm_k, tile_m, tile_n, tile_k, stage_cnt> mma_computer(
            gmem_c_local, smem_a, smem_b, smem_barrier, warp_idx, gemm_n);
        mma_computer.prepare();
        mma_computer.issue_mainloop();
        mma_computer.epi();
    }
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <typename T, int kHdIn, int kHdOut, int kTileN>
void invokeFusedAGemm(T* output, T const* mat_a, T const* mat_b, int num_tokens, hipStream_t const stream)
{
    auto const sm = tensorrt_llm::common::getSMVersion();
    if (sm < 90)
    {
        std::cerr << "FusedAGemm required CUDA ARCH >= SM_90, not supported on this architecture" << std::endl;
        assert(false);
    }
    constexpr int gemm_m = kHdOut; // 2112
    int const gemm_n = num_tokens; // 16
    constexpr int gemm_k = kHdIn;  // 7168
    constexpr int batch_size = 1;
    std::swap(mat_a, mat_b);
    constexpr int tile_m = 16;
    constexpr int tile_n = kTileN;                       // 8 or 16
    constexpr int tile_k = std::max(256, 1024 / tile_n); // 256
    constexpr int max_stage_cnt = 1024 * 192 / ((tile_m + tile_n) * tile_k * sizeof(bf16_t));
    constexpr int k_iter_cnt = gemm_k / tile_k;
    constexpr int stage_cnt
        = k_iter_cnt > max_stage_cnt ? max_stage_cnt : k_iter_cnt; // possible tunable for smallK > 1 wave n. // 22
    int cta_m_cnt = gemm_m / tile_m;
    int cta_n_cnt = (gemm_n + tile_n - 1) / tile_n;
    constexpr int barrier_bytes = (stage_cnt * 16 + 1023) / 1024 * 1024; // 4096
    constexpr int smem_bytes = ((tile_m * 2 + tile_n * 2) * tile_k * stage_cnt + barrier_bytes + 1023) / 1024 * 1024;

    dim3 grid(cta_m_cnt, cta_n_cnt, 1);
    dim3 block_size(256);
    hipLaunchConfig_t config;
    config.gridDim = grid;
    config.blockDim = block_size;
    config.dynamicSmemBytes = smem_bytes;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    if (smem_bytes >= (48 * 1024))
    {
        TLLM_CUDA_CHECK(
            hipFuncSetAttribute(reinterpret_cast<const void*>(fused_a_gemm_kernel<batch_size), gemm_m, gemm_k, tile_m, tile_n, tile_k, stage_cnt>,
                hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes));
    }
    TLLM_CUDA_CHECK(
        cudaLaunchKernelEx(&config, fused_a_gemm_kernel<batch_size, gemm_m, gemm_k, tile_m, tile_n, tile_k, stage_cnt>,
            output, mat_a, mat_b, gemm_n));
}

template void invokeFusedAGemm<__hip_bfloat16, 7168, 2112, 8>(
    __hip_bfloat16*, __hip_bfloat16 const*, __hip_bfloat16 const*, int num_tokens, hipStream_t);

template void invokeFusedAGemm<__hip_bfloat16, 7168, 2112, 16>(
    __hip_bfloat16*, __hip_bfloat16 const*, __hip_bfloat16 const*, int num_tokens, hipStream_t);
} // namespace tensorrt_llm::kernels::dsv3MinLatencyKernels
