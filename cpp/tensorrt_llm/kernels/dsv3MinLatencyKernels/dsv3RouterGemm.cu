#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/kernels/dsv3MinLatencyKernels/dsv3RouterGemm.h"
using namespace tensorrt_llm::common;

namespace tensorrt_llm::kernels::dsv3MinLatencyKernels
{

// Custom FMA implementation using PTX assembly instructions
__device__ __forceinline__ void fma(float2& d, float2 const& a, float2 const& b, float2 const& c)
{
    asm volatile("fma.rn.f32x2 %0, %1, %2, %3;\n"
                 : "=l"(reinterpret_cast<uint64_t&>(d))
                 : "l"(reinterpret_cast<uint64_t const&>(a)), "l"(reinterpret_cast<uint64_t const&>(b)),
                 "l"(reinterpret_cast<uint64_t const&>(c)));
}

// Convert 8 bfloat16 values from a uint4 to float array - optimized conversion
template <int VPT>
__device__ __forceinline__ void bf16_uint4_to_float8(uint4 const& vec, float* dst)
{
    __hip_bfloat16* bf16_ptr = reinterpret_cast<__hip_bfloat16*>(const_cast<uint4*>(&vec));

#pragma unroll
    for (int i = 0; i < VPT; i++)
    {
        dst[i] = __bfloat162float(bf16_ptr[i]);
    }
}

template <typename T, int kBlockSize, int VPT, int kNumTokens, int kNumExperts, int kHiddenDim>
__global__ __launch_bounds__(128, 1) void router_gemm_kernel(float* out, T const* mat_a, T const* mat_b)
{
    // Each block handles one expert column
    int const n_idx = blockIdx.x;
    int const tid = threadIdx.x;
    constexpr int kWarpSize = 32;
    constexpr int kNumWarps = kBlockSize / kWarpSize;
    // Constants for this kernel
    constexpr int k_elems_per_k_iteration = VPT * kBlockSize;
    constexpr int k_iterations = kHiddenDim / k_elems_per_k_iteration; // Total K iterations

    // Initialize accumulators for all M rows
    float acc[kNumTokens] = {};

    // Shared memory for warp-level reduction
    __shared__ float sm_reduction[kNumTokens][kNumWarps]; // kNumWarps

    // B matrix is in column-major order, so we can directly load a column for the n_idx expert
    T const* b_col = mat_b + n_idx * kHiddenDim;

    // Pre-compute k_base values for each iteration to help compiler optimize
    // int k_bases[k_iterations];
    int k_bases[k_iterations];
#pragma unroll
    for (int ki = 0; ki < k_iterations; ki++)
    {
        k_bases[ki] = ki * k_elems_per_k_iteration + tid * VPT;
    }

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

    // Process the GEMM in chunks
    for (int ki = 0; ki < k_iterations; ki++)
    {
        int const k_base = k_bases[ki];

        // Load B matrix values using vector load (8 bf16 values)
        uint4 b_vec = *reinterpret_cast<uint4 const*>(b_col + k_base);

        // Convert B values to float
        float b_float[VPT];
        bf16_uint4_to_float8<VPT>(b_vec, b_float);

// Process each token
#pragma unroll
        for (int m_idx = 0; m_idx < kNumTokens; m_idx++)
        {
            // Load both rows of A matrix using vector loads
            uint4 a_vec = *reinterpret_cast<uint4 const*>(mat_a + (m_idx * kHiddenDim) + k_base);

            // Convert A values to float
            float a_float[VPT];
            bf16_uint4_to_float8<VPT>(a_vec, a_float);

// Process elements in this chunk
#pragma unroll
            for (int k = 0; k < VPT; k++)
            {
                float a = a_float[k];
                float b = b_float[k];
                acc[m_idx] += a * b;
            }
        }
    }

    // Perform warp-level reduction
    int const warpSize = 32;
    int const warpId = tid / warpSize;
    int const laneId = tid % warpSize;

    // Register for warp-level reduction results
    float warp_result[kNumTokens];

#pragma unroll
    for (int m_idx = 0; m_idx < kNumTokens; m_idx++)
    {
        warp_result[m_idx] = acc[m_idx];
    }

// Perform warp-level reduction using optimized butterfly pattern
#pragma unroll
    for (int m = 0; m < kNumTokens; m++)
    {
        float sum = warp_result[m];

        // Butterfly reduction pattern
        sum += __shfl_xor_sync(0xffffffff, sum, 16);
        sum += __shfl_xor_sync(0xffffffff, sum, 8);
        sum += __shfl_xor_sync(0xffffffff, sum, 4);
        sum += __shfl_xor_sync(0xffffffff, sum, 2);
        sum += __shfl_xor_sync(0xffffffff, sum, 1);

        // Only the first thread in each warp stores to shared memory
        if (laneId == 0)
        {
            sm_reduction[m][warpId] = sum;
        }
    }

    __syncthreads();

    // Final reduction across warps (only first thread)
    if (tid == 0)
    {
#pragma unroll
        for (int m = 0; m < kNumTokens; m++)
        {
            float final_sum = 0.0f;

// Sum across the kNumWarps
#pragma unroll
            for (int w = 0; w < kNumWarps; w++)
            {
                final_sum += sm_reduction[m][w];
            }

            // Write final result
            out[m * kNumExperts + n_idx] = final_sum;
        }
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <typename T, int kNumTokens, int kNumExperts, int kHiddenDim>
void invokeRouterGemm(float* output, T const* mat_a, T const* mat_b, hipStream_t stream)
{
    constexpr int VPT = 16 / sizeof(T);
    constexpr int kBlockSize = 128;
    hipLaunchConfig_t config;
    config.gridDim = kNumExperts;
    config.blockDim = kBlockSize;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    TLLM_CUDA_CHECK(cudaLaunchKernelEx(
        &config, router_gemm_kernel<T, kBlockSize, VPT, kNumTokens, kNumExperts, kHiddenDim>, output, mat_a, mat_b));
}

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 1, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 2, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 3, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 4, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 5, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 6, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 7, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 8, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 9, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 10, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 11, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 12, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 13, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 14, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 15, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

template void tensorrt_llm::kernels::dsv3MinLatencyKernels::invokeRouterGemm<__hip_bfloat16, 16, 256, 7168>(
    float*, __hip_bfloat16 const*, __hip_bfloat16 const*, hipStream_t);

} // namespace tensorrt_llm::kernels::dsv3MinLatencyKernels
