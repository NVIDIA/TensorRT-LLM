#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/communicationKernels/moeAllReduceFusionKernels.h"
#include "tensorrt_llm/kernels/quantization.cuh"
#include <hip/hip_cooperative_groups.h>

namespace tensorrt_llm::kernels::ar_fusion::moe
{
template <int NRanks>
struct LamportComm
{
    __device__ __forceinline__ LamportComm(void** workspace, int rank)
    {
        counter_ptr = &reinterpret_cast<int*>(workspace[NRanks * 3])[0];
        flag_ptr = &reinterpret_cast<int*>(workspace[NRanks * 3])[2];
        clear_ptr = &reinterpret_cast<int*>(workspace[NRanks * 3])[4];
        flag_value = *flag_ptr;
        int comm_size = reinterpret_cast<int*>(workspace[NRanks * 3])[3];
        clear_size = *clear_ptr;
        int data_offset = flag_value % 3;
        int clear_offset = (flag_value + 2) % 3;
        for (int r = 0; r < NRanks; ++r)
        {
            data_bufs[r] = reinterpret_cast<uint8_t*>(workspace[2 * NRanks + r]) + data_offset * comm_size;
        }
        clear_buf = reinterpret_cast<uint8_t*>(workspace[2 * NRanks + rank]) + clear_offset * comm_size;
        __syncthreads();
        if (threadIdx.x == 0)
        {
            atomicAdd(counter_ptr, 1);
        }
    }

    __device__ __forceinline__ void update(int new_clear_size)
    {
        if (blockIdx.x == 0 && threadIdx.x == 0)
        {
            while (*reinterpret_cast<int volatile*>(counter_ptr) != gridDim.x)
            {
            }
            *flag_ptr = (flag_value + 1) % 3;
            *clear_ptr = new_clear_size;
            *counter_ptr = 0;
        }
    }

    int* counter_ptr;
    int* flag_ptr;
    int* clear_ptr;
    uint8_t* data_bufs[NRanks];
    uint8_t* clear_buf;
    int clear_size;
    int flag_value;
};

template <typename DType, typename PackedType>
__device__ __forceinline__ PackedType add128(PackedType const& a, PackedType const& b)
{
    static constexpr int kMathCount = sizeof(PackedType) / sizeof(DType);
    PackedType c;
#pragma unroll
    for (int i = 0; i < kMathCount; ++i)
    {
        reinterpret_cast<DType*>(&c)[i] = reinterpret_cast<DType const*>(&a)[i] + reinterpret_cast<DType const*>(&b)[i];
    }
    return c;
}

template <typename DType, typename PackedType>
__device__ __forceinline__ PackedType rms_norm(
    PackedType const& residual, PackedType const& gamma, float const eps, int hidden_dim)
{
    static constexpr int kMathCount = sizeof(PackedType) / sizeof(DType);
    __shared__ float s_val;
    PackedType norm_out;
    cg::cluster_group cluster = cg::this_cluster();
    float acc = 0.f;
#pragma unroll
    for (int i = 0; i < kMathCount; ++i)
    {
        float v = static_cast<float>(reinterpret_cast<DType const*>(&residual)[i]);
        acc += v * v;
    }
    tensorrt_llm::common::blockReduceSumV2<float, 1>(&acc);
    if (cluster.num_blocks() > 1)
    {
        if (threadIdx.x == 0)
        {
            s_val = acc;
            acc = 0.f;
        }
        cluster.sync();
        if (threadIdx.x == 0)
        {
            for (int i = 0; i < cluster.num_blocks(); ++i)
            {
                acc += *cluster.map_shared_rank(&s_val, i);
            }
        }
        cluster.sync();
    }
    if (threadIdx.x == 0)
    {
        s_val = rsqrtf(acc / hidden_dim + eps);
    }
    __syncthreads();
#pragma unroll
    for (int i = 0; i < kMathCount; ++i)
    {
        reinterpret_cast<DType*>(&norm_out)[i]
            = static_cast<DType>(static_cast<float>(reinterpret_cast<DType const*>(&residual)[i]) * s_val
                * static_cast<float>(reinterpret_cast<DType const*>(&gamma)[i]));
    }
    return norm_out;
}

template <bool ResidualOut, bool NormOut, bool QuantOut, typename DType, typename PackedType>
__device__ __forceinline__ void fused_op(
    PackedType const& val, int access_id, int token_id, int access_id_in_token, AllReduceFusionParams& params)
{
    float4 residual_val = reinterpret_cast<float4*>(params.residual_in)[access_id];
    float4 gamma_val = reinterpret_cast<float4*>(params.rms_gamma)[access_id_in_token];
    residual_val = add128<DType>(val, residual_val);
    if constexpr (ResidualOut)
    {
        reinterpret_cast<float4*>(params.residual_out)[access_id] = residual_val;
    }
    float4 norm_val = rms_norm<DType>(residual_val, gamma_val, params.rms_eps, params.hidden_dim);
    if constexpr (NormOut)
    {
        reinterpret_cast<float4*>(params.norm_out)[access_id] = norm_val;
    }
    if constexpr (QuantOut)
    {
        PackedVec<DType> pack_val = *reinterpret_cast<PackedVec<DType> const*>(&norm_val);
        auto sf_out = cvt_quant_to_fp4_get_sf_out_offset<uint32_t, 2>(std::nullopt /* batchIdx */, token_id,
            access_id_in_token, std::nullopt /* numRows */, params.hidden_dim,
            reinterpret_cast<uint32_t*>(params.scale_out), params.layout);
        reinterpret_cast<uint32_t*>(params.quant_out)[access_id]
            = cvt_warp_fp16_to_fp4(pack_val, *params.scale_factor, sf_out);
    }
}

__device__ __forceinline__ bool is_neg_zero(float v)
{
    return *reinterpret_cast<uint32_t*>(&v) == 0x80000000;
}

__device__ __forceinline__ bool is_neg_zero(float4 v)
{
    return is_neg_zero(v.x) || is_neg_zero(v.y) || is_neg_zero(v.z) || is_neg_zero(v.w);
}

__device__ __forceinline__ float4 get_neg_zero()
{
    float4 vec;
#pragma unroll
    for (int i = 0; i < 4; ++i)
    {
        reinterpret_cast<uint32_t*>(&vec)[i] = 0x80000000;
    }
    return vec;
}

__device__ __forceinline__ float4 ld_global_volatile(float4* addr)
{
    float4 val;
    asm volatile("ld.volatile.global.v4.f32 {%0, %1, %2, %3}, [%4];"
                 : "=f"(val.x), "=f"(val.y), "=f"(val.z), "=f"(val.w)
                 : "l"(addr));
    return val;
}

int get_sm_count()
{
    static int sm_count = 0;
    if (sm_count == 0)
    {
        int device_id;
        TLLM_CUDA_CHECK(hipGetDevice(&device_id));
        hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, device_id);
        sm_count = device_prop.multiProcessorCount;
    }
    return sm_count;
}

bool use_oneshot(int token_num)
{
    return token_num <= kOneShotMaxToken;
}

/////////////////////////////////////////////////////////////////
//                  * MoE Reduction Fusion *                   //
/////////////////////////////////////////////////////////////////

template <typename DType, int NRanks, bool ResidualOut, bool NormOut, bool QuantOut>
__global__ void moereduce_allreduce_fusion_kernel_oneshot_lamport(MoeReductionAllReduceFusionParams params)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    namespace cg = cooperative_groups;
    cg::cluster_group cluster = cg::this_cluster();
    cg::grid_group grid = cg::this_grid();

    // Each token is handled by one cluster
    // which token is handled by current cluster
    int token_id = grid.cluster_rank();
    // total number of token
    int num_token = params.size / params.hidden_dim;
    // Each thread handle kElemsPerAccess num elem in token. Total cluster.num_threads() to handle one token
    // For current token, which kElemsPerAccess is handled by current thread (in unit of kElemsPerAccess)
    int access_id_in_token = cluster.thread_rank();
    // Across all token, which kElemsPerAccess is handled by current thread (in unit of kElemsPerAccess)
    int access_id = token_id * params.hidden_dim / kElemsPerAccess + access_id_in_token;
    // Persistent kernel
    // stride to next token handled by current cta
    int token_stride = grid.num_clusters();
    // stride in unit of kElemsPerAccess
    int access_stride = token_stride * params.hidden_dim / kElemsPerAccess;
    // Total number of access in unit of kElemsPerAccess to handle (token_num * hidden_dim)
    // This is within one rank
    int tot_access = params.size / kElemsPerAccess;
    float4 clear_vec = get_neg_zero();

    cudaGridDependencySynchronize();
    LamportComm<NRanks> comm(params.workspace, params.rank);
    int clear_access = comm.clear_size / kElemsPerAccess;

    // * MoE related
    int threadid_in_cluster = cluster.thread_rank();
    // Start Offset within one token's hidden_size of element
    // Current thread handle token[thread_offset_within_token : thread_offset_within_token + kElemsPerAccess]
    int thread_offset_within_token = threadid_in_cluster * kElemsPerAccess;

    union ACC_TYPE
    {
        float4 packed;
        DType unpacked[kElemsPerAccess];
    };

    // Persistent Kernel
    // Each cluster iterate through all token it need to handle
    for (int token_id = grid.cluster_rank(); token_id < num_token; token_id += grid.num_clusters())
    {
        if (thread_offset_within_token >= params.hidden_dim)
        {
            break;
        }

        // * MoE Reduce
        // Offset within (num_token, hidden_size) in unit of element
        int thread_offset_across_token = token_id * params.hidden_dim + thread_offset_within_token;

        ACC_TYPE accumulator;
#pragma unroll
        for (int i = 0; i < kElemsPerAccess; ++i)
        {
            accumulator.unpacked[i] = static_cast<DType>(0);
        }

        // * Iterate through all active expert
        int num_actexp = *(params.moe_reduction_device_num_experts);
        for (int actexp_i = 0; actexp_i < num_actexp; ++actexp_i)
        {
            // * Load active expert i's token j's partial data
            // Offset within (num_act_exp, num_token, hidden_size) in unit of element
            int thread_offset_across_actexp_token
                = actexp_i * (params.hidden_dim * num_token) + thread_offset_across_token;
            ACC_TYPE actexp_i_data;
            actexp_i_data.packed = reinterpret_cast<float4 const*>(
                params.moe_reduction_active_experts_token_input)[thread_offset_across_actexp_token / kElemsPerAccess];

            // * Load active expert i's token j's scale
            int thread_offset_scale = actexp_i * num_token + token_id;
            float actexp_i_token_j_scale
                = reinterpret_cast<float const*>(params.moe_reduction_scale_input)[thread_offset_scale];

            // * acc += scale(data)
#pragma unroll
            for (int i = 0; i < kElemsPerAccess; ++i)
            {
                // assume computation is done in ScaleType
                accumulator.unpacked[i]
                    += static_cast<DType>((static_cast<float>(actexp_i_data.unpacked[i]) * actexp_i_token_j_scale));
            }
        }

        // * FC2 + reduced(gGEMM2)
        ACC_TYPE fc2_data;
        fc2_data.packed = reinterpret_cast<float4 const*>(
            params.moe_reduction_token_input)[thread_offset_across_token / kElemsPerAccess];
#pragma unroll
        for (int i = 0; i < kElemsPerAccess; ++i)
        {
            accumulator.unpacked[i] += fc2_data.unpacked[i];
        }

        // * AR Store
        int access_id = token_id * params.hidden_dim / kElemsPerAccess + access_id_in_token;
        int idx = access_id;
        float val[4] = {accumulator.packed.x, accumulator.packed.y, accumulator.packed.z, accumulator.packed.w};

#pragma unroll
        for (int i = 0; i < 4; ++i)
        {
            // Handle two bf16/fp16 at one time
            if (is_neg_zero(val[i]))
            {
                val[i] = 0.f;
            }
        }
#pragma unroll
        for (int r = 0; r < NRanks; ++r)
        {
            // STG.128 to remote rank
            reinterpret_cast<float4*>(comm.data_bufs[r])[params.rank * tot_access + idx]
                = *reinterpret_cast<float4*>(val);
        }
    }

    // * Clear previous buffer
    for (int idx = access_id; idx < clear_access; idx += access_stride)
    {
        reinterpret_cast<float4*>(comm.clear_buf)[idx] = clear_vec;
    }

    // * AR Load + Fusion
    for (int idx = access_id, tidx = token_id; idx < tot_access; idx += access_stride, tidx += token_stride)
    {
        // * AR Load
        float4 vals[NRanks];
        bool done = false;
        while (!done)
        {
            done = true;
#pragma unroll
            for (int r = 0; r < NRanks; ++r)
            {
                // LDG.128 from local rank
                vals[r]
                    = ld_global_volatile(&reinterpret_cast<float4*>(comm.data_bufs[params.rank])[r * tot_access + idx]);
                done &= !is_neg_zero(vals[r]);
            }
        }
        float4 sum_val = vals[0];
#pragma unroll
        for (int r = 1; r < NRanks; ++r)
        {
            sum_val = add128<DType>(sum_val, vals[r]);
        }

        // * Fuse
        fused_op<ResidualOut, NormOut, QuantOut, DType>(sum_val, idx, tidx, access_id_in_token, params);
    }
    comm.update(params.size * NRanks);
    cudaTriggerProgrammaticLaunchCompletion();
#endif

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <typename DType, int NRanks, bool ResidualOut, bool NormOut, bool QuantOut>
void launch_oneshot_moereduce_lamport(MoeReductionAllReduceFusionParams const& params, hipLaunchConfig_t& cfg)
{
    TLLM_CUDA_CHECK(cudaLaunchKernelEx(&cfg,
        moereduce_allreduce_fusion_kernel_oneshot_lamport<DType, NRanks, ResidualOut, NormOut, QuantOut>, params));
}

template <typename DType, int NRanks, bool ResidualOut, bool NormOut, bool QuantOut>
void moereduction_allreduce_fusion_kernel_launcher(MoeReductionAllReduceFusionParams const& params)
{
    int token_num = params.size / params.hidden_dim;
    bool oneshot = use_oneshot(token_num);
    // Only support one shot
    TLLM_CHECK(oneshot);
    // Each token is handled by one cluster
    int cluster_num = token_num;
    // Total number of threads (within one cluster) that's need to handle one token
    // given that each thread handle kElemsPerAccess
    int threads_per_token = params.hidden_dim / kElemsPerAccess;
    // Total number of warp (within one cluster) that's need to handle one token
    // given that each thread handle kElemsPerAccess
    int warps_per_token = (threads_per_token + 31) / 32;
    int cluster_size = 8;
    while (warps_per_token % cluster_size != 0)
    {
        cluster_size /= 2;
    }
    int block_size = warps_per_token / cluster_size * 32;
    TLLM_CHECK(block_size <= 1024 && cluster_size > 0);
    int sm_count = get_sm_count();
    int grid_size = (std::min(sm_count, cluster_num * cluster_size) / cluster_size) * cluster_size;
    hipLaunchConfig_t cfg;
    hipLaunchAttribute attribute[2];
    cfg.gridDim = grid_size;
    cfg.blockDim = block_size;
    cfg.dynamicSmemBytes = 0;
    cfg.stream = params.stream;
    attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attribute[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL() ? 1 : 0;
    attribute[1].id = cudaLaunchAttributeClusterDimension;
    attribute[1].val.clusterDim.x = cluster_size;
    attribute[1].val.clusterDim.y = 1;
    attribute[1].val.clusterDim.z = 1;
    cfg.attrs = attribute;
    cfg.numAttrs = 2;
    if (oneshot)
    {
        launch_oneshot_moereduce_lamport<DType, NRanks, ResidualOut, NormOut, QuantOut>(params, cfg);
    }
}

void moereduction_allreduce_fusion_op(MoeReductionAllReduceFusionParams const& params)
{
#define MOE_DISPATCH1(DTYPE, NRANKS, RESIDUAL_OUT, NORM_OUT, QUANT_OUT)                                                \
    return moereduction_allreduce_fusion_kernel_launcher<DTYPE, NRANKS, RESIDUAL_OUT, NORM_OUT, QUANT_OUT>(params);
#define MOE_DISPATCH0(NRANKS, RESIDUAL_OUT, NORM_OUT, QUANT_OUT)                                                       \
    if (params.nranks == NRANKS && params.dtype == nvinfer1::DataType::kHALF)                                          \
    {                                                                                                                  \
        MOE_DISPATCH1(half, NRANKS, RESIDUAL_OUT, NORM_OUT, QUANT_OUT);                                                \
    }                                                                                                                  \
    else if (params.nranks == NRANKS && params.dtype == nvinfer1::DataType::kBF16)                                     \
    {                                                                                                                  \
        MOE_DISPATCH1(__hip_bfloat16, NRANKS, RESIDUAL_OUT, NORM_OUT, QUANT_OUT);                                       \
    }

    TLLM_CHECK(params.residual_in && params.rms_gamma);
    TLLM_CHECK(params.moe_reduction_scale_input && params.moe_reduction_active_experts_token_input
        && params.moe_reduction_token_input);
    TLLM_CHECK(params.size % params.hidden_dim == 0);
    TLLM_CHECK(params.hidden_dim % kElemsPerAccess == 0);
    if (params.residual_out && not params.norm_out && params.quant_out)
    {
        // pattern1: AR+Add_RMS+Quant
        // [m, 7168] bf16 allreduce_in, [m, 7168] bf16 residual_in
        // [m, 7168] bf16 residual_out, [m, 7168] fp4 quant_out
        MOE_DISPATCH0(2, true, false, true);
        MOE_DISPATCH0(4, true, false, true);
        MOE_DISPATCH0(8, true, false, true);
        MOE_DISPATCH0(16, true, false, true);
    }
    else if (not params.residual_out && params.norm_out && not params.quant_out)
    {
        // pattern2: AR+AddRMS
        // [m, 7168] bf16 allreduce_in, [m, 7168] bf16 residual_in
        // [m, 7168] bf16 norm_out
        MOE_DISPATCH0(2, false, true, false);
        MOE_DISPATCH0(4, false, true, false);
        MOE_DISPATCH0(8, false, true, false);
        MOE_DISPATCH0(16, false, true, false);
    }
    else if (params.residual_out && params.norm_out && not params.quant_out)
    {
        MOE_DISPATCH0(2, true, true, false);
        MOE_DISPATCH0(4, true, true, false);
        MOE_DISPATCH0(8, true, true, false);
        MOE_DISPATCH0(16, true, true, false);
    }
    else if (params.residual_out && params.norm_out && params.quant_out)
    {
        // for test
        MOE_DISPATCH0(2, true, true, true);
        MOE_DISPATCH0(4, true, true, true);
        MOE_DISPATCH0(8, true, true, true);
        MOE_DISPATCH0(16, true, true, true);
    }
    TLLM_CHECK_WITH_INFO(false, "allreduce_fusion_kernel: unsupported pattern!");
}

}; // namespace tensorrt_llm::kernels::ar_fusion::moe
