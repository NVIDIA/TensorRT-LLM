#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/communicationKernels/allReduceFusionKernels.h"
#include "tensorrt_llm/kernels/quantization.cuh"
#include <hip/hip_cooperative_groups.h>

namespace tensorrt_llm::kernels::ar_fusion
{
template <int NRanks>
struct SyncComm
{
    __device__ __forceinline__ SyncComm(void** workspace)
    {
        counter_ptr = &reinterpret_cast<int*>(workspace[NRanks * 3])[0];
        flag_ptr = &reinterpret_cast<int*>(workspace[NRanks * 3])[1];
        flag_value = *flag_ptr;
        for (int r = 0; r < NRanks; ++r)
        {
            comm_bufs[r] = workspace[r];
            barrier_flags[r] = workspace[NRanks + r];
        }
        __syncthreads();
        if (threadIdx.x == 0)
        {
            atomicAdd(counter_ptr, 1);
        }
    }

    __device__ __forceinline__ void update(int new_flag_value)
    {
        if (blockIdx.x == 0 && threadIdx.x == 0)
        {
            while (*reinterpret_cast<int volatile*>(counter_ptr) != gridDim.x)
            {
            }
            *flag_ptr = new_flag_value;
            *counter_ptr = 0;
        }
    }

    int* counter_ptr;
    int* flag_ptr;
    void* comm_bufs[NRanks];
    void* barrier_flags[NRanks];
    int flag_value;
};

template <int NRanks>
struct LamportComm
{
    __device__ __forceinline__ LamportComm(void** workspace, int rank)
    {
        counter_ptr = &reinterpret_cast<int*>(workspace[NRanks * 3])[0];
        flag_ptr = &reinterpret_cast<int*>(workspace[NRanks * 3])[2];
        clear_ptr = &reinterpret_cast<int*>(workspace[NRanks * 3])[4];
        flag_value = *flag_ptr;
        int comm_size = reinterpret_cast<int*>(workspace[NRanks * 3])[3];
        clear_size = *clear_ptr;
        int data_offset = flag_value % 3;
        int clear_offset = (flag_value + 2) % 3;
        for (int r = 0; r < NRanks; ++r)
        {
            data_bufs[r] = reinterpret_cast<uint8_t*>(workspace[2 * NRanks + r]) + data_offset * comm_size;
        }
        clear_buf = reinterpret_cast<uint8_t*>(workspace[2 * NRanks + rank]) + clear_offset * comm_size;
        __syncthreads();
        if (threadIdx.x == 0)
        {
            atomicAdd(counter_ptr, 1);
        }
    }

    __device__ __forceinline__ void update(int new_clear_size)
    {
        if (blockIdx.x == 0 && threadIdx.x == 0)
        {
            while (*reinterpret_cast<int volatile*>(counter_ptr) != gridDim.x)
            {
            }
            *flag_ptr = (flag_value + 1) % 3;
            *clear_ptr = new_clear_size;
            *counter_ptr = 0;
        }
    }

    int* counter_ptr;
    int* flag_ptr;
    int* clear_ptr;
    uint8_t* data_bufs[NRanks];
    uint8_t* clear_buf;
    int clear_size;
    int flag_value;
};

template <int NRanks>
class Barrier
{
public:
    __device__ __forceinline__ Barrier(int rank, SyncComm<NRanks> const& comm)
    {
        if (threadIdx.x < NRanks)
        {
            m_flag_value = comm.flag_value;
            int current_rank = rank;
            int target_rank = threadIdx.x;
            m_target_flag = reinterpret_cast<int*>(comm.barrier_flags[target_rank]) + current_rank;
            m_current_flag
                = reinterpret_cast<int*>(comm.barrier_flags[current_rank]) + blockIdx.x * NRanks + target_rank;
        }
    }

    __device__ __forceinline__ void sync()
    {
        __syncthreads();
        if (threadIdx.x < NRanks)
        {
            m_flag_value = next_flag(m_flag_value);
            // To avoid the ABA problem, we need to synchronize the correct flag value to all barrier_flags, even if the
            // corresponding CTA has not been launched.
            for (int flag_idx = blockIdx.x; flag_idx < kBarrierFlagCount; flag_idx += gridDim.x)
            {
                st_flag(m_target_flag + flag_idx * NRanks, m_flag_value);
            }
            while (ld_flag(m_current_flag) == prev_flag(m_flag_value))
            {
            }
        }
        __syncthreads();
    }

protected:
    __device__ __forceinline__ void st_flag(int* addr, int flag)
    {
        asm volatile("st.global.release.sys.b32 [%1], %0;" ::"r"(flag), "l"(addr));
    }

    __device__ __forceinline__ int ld_flag(int* addr)
    {
        int flag;
        asm volatile("ld.global.acquire.sys.b32 %0, [%1];" : "=r"(flag) : "l"(addr));
        return flag;
    }

    __device__ __forceinline__ int next_flag(int flag)
    {
        return flag == 2 ? 0 : flag + 1;
    }

    __device__ __forceinline__ int prev_flag(int flag)
    {
        return flag == 0 ? 2 : flag - 1;
    }

public:
    int m_flag_value;

private:
    int* m_target_flag;
    int* m_current_flag;
};

template <typename DType, typename PackedType>
__device__ __forceinline__ PackedType add128(PackedType const& a, PackedType const& b)
{
    static constexpr int kMathCount = sizeof(PackedType) / sizeof(DType);
    PackedType c;
#pragma unroll
    for (int i = 0; i < kMathCount; ++i)
    {
        reinterpret_cast<DType*>(&c)[i] = reinterpret_cast<DType const*>(&a)[i] + reinterpret_cast<DType const*>(&b)[i];
    }
    return c;
}

template <AllReduceFusionPattern Pattern, typename DType>
class FusedOp
{
    static constexpr int kMathCount = sizeof(float4) / sizeof(DType);

public:
    __device__ __forceinline__ FusedOp(AllReduceFusionParams const& params, int access_id, int access_id_in_token)
        : m_params(params)
        , m_access_id(access_id)
        , m_access_id_in_token(access_id_in_token)
    {
        if constexpr (HasRMSNorm<Pattern>)
        {
            m_gamma_val = reinterpret_cast<float4*>(params.rms_gamma)[m_access_id_in_token];
        }
        if constexpr (HasResidual<Pattern>)
        {
            m_residual_val = reinterpret_cast<float4*>(params.residual_in)[m_access_id];
        }
        if constexpr (GetQuantType<Pattern> == QuantType::kFP8)
        {
            m_scale_factor = 1.f / *params.scale_factor;
        }
        else if constexpr (GetQuantType<Pattern> == QuantType::kFP4)
        {
            m_scale_factor = *params.scale_factor;
        }
    }

    __device__ __forceinline__ void update(int access_id)
    {

        if (m_access_id != access_id)
        {
            m_access_id = access_id;
            if constexpr (HasResidual<Pattern>)
            {
                m_residual_val = reinterpret_cast<float4*>(m_params.residual_in)[m_access_id];
            }
        }
    }

    __device__ __forceinline__ void operator()(float4 val, int token_id)
    {
        if constexpr (HasAllReduceOut<Pattern>)
        {
            reinterpret_cast<float4*>(m_params.allreduce_out)[m_access_id] = val;
        }
        if constexpr (HasResidual<Pattern>)
        {
            val = add128<DType>(val, m_residual_val);
            if constexpr (HasResidualOut<Pattern>)
            {
                reinterpret_cast<float4*>(m_params.residual_out)[m_access_id] = val;
            }
        }
        if constexpr (HasRMSNorm<Pattern>)
        {
            val = rms_norm(val, m_gamma_val);
            if constexpr (HasNormOut<Pattern>)
            {
                reinterpret_cast<float4*>(m_params.norm_out)[m_access_id] = val;
            }
        }
        if constexpr (GetQuantType<Pattern> == QuantType::kFP4)
        {
            PackedVec<DType> pack_val = *reinterpret_cast<PackedVec<DType> const*>(&val);
            auto sf_out = cvt_quant_to_fp4_get_sf_out_offset<uint32_t, 2>(std::nullopt, token_id, m_access_id_in_token,
                std::nullopt, m_params.hidden_dim, reinterpret_cast<uint32_t*>(m_params.scale_out), m_params.layout);
            reinterpret_cast<uint32_t*>(m_params.quant_out)[m_access_id]
                = cvt_warp_fp16_to_fp4(pack_val, m_scale_factor, sf_out);
        }
        else if constexpr (GetQuantType<Pattern> == QuantType::kFP8)
        {
            using PackedQuantizedType = std::conditional_t<std::is_same_v<DType, float>, float, float2>;
            PackedQuantizedType ret;
#pragma unroll
            for (int i = 0; i < kMathCount; ++i)
            {
                reinterpret_cast<__hip_fp8_e4m3_fnuz*>(&ret)[i] = static_cast<__hip_fp8_e4m3_fnuz>(
                    static_cast<float>(reinterpret_cast<DType*>(&val)[i]) * m_scale_factor);
            }
            reinterpret_cast<PackedQuantizedType*>(m_params.quant_out)[m_access_id] = ret;
        }
        else
        {
            static_assert(GetQuantType<Pattern> == QuantType::kNone, "Invalid quant type");
        }
    }

protected:
    __device__ __forceinline__ float4 rms_norm(float4 const& residual, float4 const& gamma)
    {
        __shared__ float s_val;
        float4 norm_out;
        float acc = 0.f;
#pragma unroll
        for (int i = 0; i < kMathCount; ++i)
        {
            float v = static_cast<float>(reinterpret_cast<DType const*>(&residual)[i]);
            acc += v * v;
        }
        tensorrt_llm::common::blockReduceSumV2<float, 1>(&acc);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        cg::cluster_group cluster = cg::this_cluster();
        if (cluster.num_blocks() > 1)
        {
            if (threadIdx.x == 0)
            {
                s_val = acc;
                acc = 0.f;
            }
            cluster.sync();
            if (threadIdx.x == 0)
            {
                for (int i = 0; i < cluster.num_blocks(); ++i)
                {
                    acc += *cluster.map_shared_rank(&s_val, i);
                }
            }
            cluster.sync();
        }
#endif
        if (threadIdx.x == 0)
        {
            s_val = rsqrtf(acc / m_params.hidden_dim + m_params.rms_eps);
        }
        __syncthreads();
#pragma unroll
        for (int i = 0; i < kMathCount; ++i)
        {
            reinterpret_cast<DType*>(&norm_out)[i]
                = static_cast<DType>(static_cast<float>(reinterpret_cast<DType const*>(&residual)[i]) * s_val
                    * static_cast<float>(reinterpret_cast<DType const*>(&gamma)[i]));
        }
        return norm_out;
    }

private:
    AllReduceFusionParams const& m_params;
    int m_access_id;
    int m_access_id_in_token;
    float m_scale_factor;
    float4 m_residual_val;
    float4 m_gamma_val;
};

__device__ __forceinline__ bool is_neg_zero(float v)
{
    return *reinterpret_cast<uint32_t*>(&v) == 0x80000000;
}

__device__ __forceinline__ bool is_neg_zero(float4 v)
{
    return is_neg_zero(v.x) || is_neg_zero(v.y) || is_neg_zero(v.z) || is_neg_zero(v.w);
}

__device__ __forceinline__ float4 get_neg_zero()
{
    float4 vec;
#pragma unroll
    for (int i = 0; i < 4; ++i)
    {
        reinterpret_cast<uint32_t*>(&vec)[i] = 0x80000000;
    }
    return vec;
}

__device__ __forceinline__ float4 ld_global_volatile(float4* addr)
{
    float4 val;
    asm volatile("ld.volatile.global.v4.f32 {%0, %1, %2, %3}, [%4];"
                 : "=f"(val.x), "=f"(val.y), "=f"(val.z), "=f"(val.w)
                 : "l"(addr));
    return val;
}

template <typename DType, int NRanks, bool Fp32Acc>
__device__ __forceinline__ float4 allreduce_sum(float4* vals)
{
    if constexpr (Fp32Acc)
    {
        static_assert(!std::is_same_v<DType, float>);
        float acc_f32[kElemsPerAccess<DType>];
#pragma unroll
        for (int i = 0; i < kElemsPerAccess<DType>; ++i)
        {
            acc_f32[i] = static_cast<float>(reinterpret_cast<DType*>(&vals[0])[i]);
        }
#pragma unroll
        for (int r = 1; r < NRanks; ++r)
        {
#pragma unroll
            for (int i = 0; i < kElemsPerAccess<DType>; ++i)
            {
                acc_f32[i] += static_cast<float>(reinterpret_cast<DType*>(&vals[r])[i]);
            }
        }
        float4 acc;
#pragma unroll
        for (int i = 0; i < kElemsPerAccess<DType>; ++i)
        {
            reinterpret_cast<DType*>(&acc)[i] = static_cast<DType>(acc_f32[i]);
        }
        return acc;
    }
    else
    {
        float4 acc = vals[0];
#pragma unroll
        for (int r = 1; r < NRanks; ++r)
        {
            acc = add128<DType>(acc, vals[r]);
        }
        return acc;
    }
}

template <typename DType>
class IndexHelper
{
public:
    __device__ __forceinline__ IndexHelper(AllReduceFusionParams const& params)
    {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        namespace cg = cooperative_groups;
        cg::cluster_group cluster = cg::this_cluster();
        cg::grid_group grid = cg::this_grid();
        token_id = grid.cluster_rank();
        access_id_in_token = cluster.thread_rank();
        token_stride = grid.num_clusters();
#else
        token_id = blockIdx.x;
        access_id_in_token = threadIdx.x;
        token_stride = gridDim.x;
#endif
        access_id = token_id * params.hidden_dim / kElemsPerAccess<DType> + access_id_in_token;
        access_stride = token_stride * params.hidden_dim / kElemsPerAccess<DType>;
        tot_access = params.size / kElemsPerAccess<DType>;
    }

    int token_id;
    int access_id_in_token;
    int token_stride;
    int access_id;
    int access_stride;
    int tot_access;
};

template <AllReduceFusionPattern Pattern, typename DType, int NRanks, bool Fp32Acc>
__global__ void allreduce_fusion_kernel_oneshot_lamport(AllReduceFusionParams params)
{
    IndexHelper<DType> index_helper(params);
    int token_id = index_helper.token_id;
    int access_id_in_token = index_helper.access_id_in_token;
    int token_stride = index_helper.token_stride;
    int access_id = index_helper.access_id;
    int access_stride = index_helper.access_stride;
    int tot_access = index_helper.tot_access;
    float4 clear_vec = get_neg_zero();
    FusedOp<Pattern, DType> fused_op(params, access_id, access_id_in_token);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif
    LamportComm<NRanks> comm(params.workspace, params.rank);
    int clear_access = comm.clear_size / kElemsPerAccess<DType>;

    for (int idx = access_id; idx < tot_access; idx += access_stride)
    {
        float val[4];
        *reinterpret_cast<float4*>(val) = reinterpret_cast<float4*>(params.allreduce_in)[idx];
#pragma unroll
        for (int i = 0; i < kElemsPerAccess<DType> / sizeof(float); ++i)
        {
            if (is_neg_zero(val[i]))
            {
                val[i] = 0.f;
            }
        }
#pragma unroll
        for (int r = 0; r < NRanks; ++r)
        {
            // Push data to other ranks
            reinterpret_cast<float4*>(comm.data_bufs[r])[params.rank * tot_access + idx]
                = *reinterpret_cast<float4*>(val);
        }
    }
    for (int idx = access_id; idx < clear_access; idx += access_stride)
    {
        // Clear comm buffer that previous kernel used
        reinterpret_cast<float4*>(comm.clear_buf)[idx] = clear_vec;
    }

    for (int idx = access_id, tidx = token_id; idx < tot_access; idx += access_stride, tidx += token_stride)
    {
        fused_op.update(idx);
        float4 vals[NRanks];
        bool done = false;
        while (!done)
        {
            done = true;
#pragma unroll
            for (int r = 0; r < NRanks; ++r)
            {
                // LDG.128 from local rank
                vals[r]
                    = ld_global_volatile(&reinterpret_cast<float4*>(comm.data_bufs[params.rank])[r * tot_access + idx]);
                done &= !is_neg_zero(vals[r]);
            }
        }
        float4 sum_val = allreduce_sum<DType, NRanks, Fp32Acc>(vals);
        fused_op(sum_val, tidx);
    }
    comm.update(params.size * NRanks);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

template <AllReduceFusionPattern Pattern, typename DType, int NRanks, bool Fp32Acc>
__global__ void allreduce_fusion_kernel_twoshot_sync(
    AllReduceFusionParams params, std::array<int, NRanks> begin_tokens, std::array<int, NRanks> token_num_per_ranks)
{
    IndexHelper<DType> index_helper(params);
    int token_id = index_helper.token_id;
    int access_id_in_token = index_helper.access_id_in_token;
    int token_stride = index_helper.token_stride;
    int access_id = index_helper.access_id;
    int access_stride = index_helper.access_stride;
    int tot_access = index_helper.tot_access;
    FusedOp<Pattern, DType> fused_op(params, access_id, access_id_in_token);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif
    SyncComm<NRanks> comm(params.workspace);
#pragma unroll
    for (int r = 0; r < NRanks; ++r)
    {
        int comm_access_id = access_id + begin_tokens[r] * params.hidden_dim / kElemsPerAccess<DType>;
        int comm_tot_access = (begin_tokens[r] + token_num_per_ranks[r]) * params.hidden_dim / kElemsPerAccess<DType>;
        for (int idx = comm_access_id; idx < comm_tot_access; idx += access_stride)
        {
            reinterpret_cast<float4*>(comm.comm_bufs[params.rank])[idx]
                = reinterpret_cast<float4*>(params.allreduce_in)[idx];
        }
    }
    Barrier<NRanks> barrier(params.rank, comm);
    barrier.sync();
    int comm_access_id = access_id + begin_tokens[params.rank] * params.hidden_dim / kElemsPerAccess<DType>;
    int comm_tot_access
        = (begin_tokens[params.rank] + token_num_per_ranks[params.rank]) * params.hidden_dim / kElemsPerAccess<DType>;
    for (int idx = comm_access_id; idx < comm_tot_access; idx += access_stride)
    {
        float4 vals[NRanks];
#pragma unroll
        for (int r = 0; r < NRanks; ++r)
        {
            vals[r] = reinterpret_cast<float4*>(comm.comm_bufs[r])[idx];
        }
        float4 sum_val = allreduce_sum<DType, NRanks, Fp32Acc>(vals);
#pragma unroll
        for (int r = 0; r < NRanks; ++r)
        {
            reinterpret_cast<float4*>(comm.comm_bufs[r])[tot_access + idx] = sum_val;
        }
    }
    barrier.sync();
#pragma unroll
    for (int r = 0; r < NRanks; ++r)
    {
        int comm_access_id = access_id + begin_tokens[r] * params.hidden_dim / kElemsPerAccess<DType>;
        int comm_token_id = token_id + begin_tokens[r];
        int comm_tot_access = (begin_tokens[r] + token_num_per_ranks[r]) * params.hidden_dim / kElemsPerAccess<DType>;
        for (int idx = comm_access_id, tidx = comm_token_id; idx < comm_tot_access;
             idx += access_stride, tidx += token_stride)
        {
            fused_op.update(idx);
            float4 sum_val = reinterpret_cast<float4*>(comm.comm_bufs[params.rank])[tot_access + idx];
            fused_op(sum_val, tidx);
        }
    }
    comm.update(barrier.m_flag_value);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
}

int get_sm_count()
{
    static int sm_count = 0;
    if (sm_count == 0)
    {
        int device_id;
        TLLM_CUDA_CHECK(hipGetDevice(&device_id));
        hipDeviceProp_t device_prop;
        hipGetDeviceProperties(&device_prop, device_id);
        sm_count = device_prop.multiProcessorCount;
    }
    return sm_count;
}

template <AllReduceFusionPattern Pattern, typename DType, int NRanks, bool Fp32Acc>
void launch_oneshot_lamport(AllReduceFusionParams const& params, hipLaunchConfig_t& cfg)
{
    TLLM_CUDA_CHECK(
        cudaLaunchKernelEx(&cfg, allreduce_fusion_kernel_oneshot_lamport<Pattern, DType, NRanks, Fp32Acc>, params));
}

template <AllReduceFusionPattern Pattern, typename DType, int NRanks, bool Fp32Acc>
void launch_twoshot_sync(AllReduceFusionParams const& params, hipLaunchConfig_t& cfg,
    std::array<int, NRanks> begin_tokens, std::array<int, NRanks> token_num_per_ranks)
{
    TLLM_CUDA_CHECK(cudaLaunchKernelEx(&cfg, allreduce_fusion_kernel_twoshot_sync<Pattern, DType, NRanks, Fp32Acc>,
        params, begin_tokens, token_num_per_ranks));
}

bool use_oneshot(int token_num)
{
    return token_num <= kOneShotMaxToken;
}

template <AllReduceFusionPattern Pattern, typename DType, int NRanks, bool Fp32Acc>
void allreduce_fusion_kernel_launcher(AllReduceFusionParams const& params)
{
    static int SM = tensorrt_llm::common::getSMVersion();
    int token_num = params.size / params.hidden_dim;
    bool oneshot = use_oneshot(token_num);
    int cluster_num = token_num;
    std::array<int, NRanks> begin_tokens, token_num_per_ranks;
    if (!oneshot)
    {
        int remaining_token = token_num % NRanks;
        int token_num_per_rank = token_num / NRanks;
        cluster_num = token_num_per_rank;
        if (remaining_token)
        {
            cluster_num++;
        }
        for (int r = 0; r < NRanks; ++r)
        {
            begin_tokens[r] = r * token_num_per_rank + (remaining_token > r ? r : remaining_token);
            token_num_per_ranks[r] = token_num_per_rank + (remaining_token > r ? 1 : 0);
        }
    }
    int threads_per_token = params.hidden_dim / kElemsPerAccess<DType>;
    int warps_per_token = (threads_per_token + 31) / 32;
    int cluster_size;
    if (SM >= 90)
    {
        cluster_size = 8;
    }
    else
    {
        cluster_size = 1;
    }
    while (warps_per_token % cluster_size != 0 && cluster_size > 1)
    {
        cluster_size /= 2;
    }
    int warps_per_block = warps_per_token / cluster_size;
    while (warps_per_block < 4 && cluster_size >= 2)
    {
        warps_per_block *= 2;
        cluster_size /= 2;
    }
    int block_size = warps_per_block * 32;
    TLLM_CHECK(block_size <= 1024 && cluster_size > 0);
    int sm_count = get_sm_count();
    int grid_size = (std::min(sm_count, cluster_num * cluster_size) / cluster_size) * cluster_size;
    hipLaunchConfig_t cfg;
    hipLaunchAttribute attribute[2];
    cfg.gridDim = grid_size;
    cfg.blockDim = block_size;
    cfg.dynamicSmemBytes = 0;
    cfg.stream = params.stream;
    attribute[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attribute[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL() ? 1 : 0;
    attribute[1].id = cudaLaunchAttributeClusterDimension;
    attribute[1].val.clusterDim.x = cluster_size;
    attribute[1].val.clusterDim.y = 1;
    attribute[1].val.clusterDim.z = 1;
    cfg.attrs = attribute;
    cfg.numAttrs = SM >= 90 ? 2 : 0;
    if (oneshot)
    {
        launch_oneshot_lamport<Pattern, DType, NRanks, Fp32Acc>(params, cfg);
    }
    else
    {
        launch_twoshot_sync<Pattern, DType, NRanks, Fp32Acc>(params, cfg, begin_tokens, token_num_per_ranks);
    }
}

bool use_fp32_acc()
{
    // we use fp16 acc type by default due to keep align with nccl
    static char* fp32_acc = std::getenv("ALL_REDUCE_FUSION_KERNEL_ACC_FP32");
    return fp32_acc != nullptr;
}

void allreduce_fusion_op(AllReduceFusionParams const& params)
{
#define DISPATCH_ACC_TYPE(DType, Pattern, NRanks)                                                                      \
    if constexpr (std::is_same_v<DType, float>)                                                                        \
    {                                                                                                                  \
        return allreduce_fusion_kernel_launcher<Pattern, DType, NRanks, false>(params);                                \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        if (fp32_acc)                                                                                                  \
        {                                                                                                              \
            return allreduce_fusion_kernel_launcher<Pattern, DType, NRanks, true>(params);                             \
        }                                                                                                              \
        else                                                                                                           \
        {                                                                                                              \
            return allreduce_fusion_kernel_launcher<Pattern, DType, NRanks, false>(params);                            \
        }                                                                                                              \
    }

#define DISPATCH_PATTERN(DType, NRanks)                                                                                \
    if (params.pattern == AllReduceFusionPattern::kAllReduce)                                                          \
    {                                                                                                                  \
        DISPATCH_ACC_TYPE(DType, AllReduceFusionPattern::kAllReduce, NRanks);                                          \
    }                                                                                                                  \
    else if (params.pattern == AllReduceFusionPattern::kARResidualRMSNorm)                                             \
    {                                                                                                                  \
        DISPATCH_ACC_TYPE(DType, AllReduceFusionPattern::kARResidualRMSNorm, NRanks);                                  \
    }                                                                                                                  \
    else if (params.pattern == AllReduceFusionPattern::kARResidualRMSNormFP8Quant)                                     \
    {                                                                                                                  \
        DISPATCH_ACC_TYPE(DType, AllReduceFusionPattern::kARResidualRMSNormFP8Quant, NRanks);                          \
    }                                                                                                                  \
    else if (params.pattern == AllReduceFusionPattern::kARResidualRMSNormFP4Quant)                                     \
    {                                                                                                                  \
        if constexpr (!std::is_same_v<DType, float>)                                                                   \
        {                                                                                                              \
            DISPATCH_ACC_TYPE(DType, AllReduceFusionPattern::kARResidualRMSNormFP4Quant, NRanks);                      \
        }                                                                                                              \
        else                                                                                                           \
        {                                                                                                              \
            TLLM_CHECK_WITH_INFO(false,                                                                                \
                "allreduce_fusion_kernel: AllReduceFusionPattern=kARResidualRMSNormFP4Quant can not work with "        \
                "DType=float!");                                                                                       \
        }                                                                                                              \
    }                                                                                                                  \
    else if (params.pattern == AllReduceFusionPattern::kARResidualRMSNormOutFP8Quant)                                  \
    {                                                                                                                  \
        DISPATCH_ACC_TYPE(DType, AllReduceFusionPattern::kARResidualRMSNormOutFP8Quant, NRanks);                       \
    }                                                                                                                  \
    else if (params.pattern == AllReduceFusionPattern::kARResidualRMSNormOutFP4Quant)                                  \
    {                                                                                                                  \
        if constexpr (!std::is_same_v<DType, float>)                                                                   \
        {                                                                                                              \
            DISPATCH_ACC_TYPE(DType, AllReduceFusionPattern::kARResidualRMSNormOutFP4Quant, NRanks);                   \
        }                                                                                                              \
        else                                                                                                           \
        {                                                                                                              \
            TLLM_CHECK_WITH_INFO(false,                                                                                \
                "allreduce_fusion_kernel: AllReduceFusionPattern=kARResidualRMSNormOutFP4Quant can not work with "     \
                "DType=float!");                                                                                       \
        }                                                                                                              \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        TLLM_CHECK_WITH_INFO(false, "allreduce_fusion_kernel: unsupported pattern!");                                  \
    }

#define DISPATCH_DTYPE(NRanks)                                                                                         \
    if (params.dtype == nvinfer1::DataType::kHALF)                                                                     \
    {                                                                                                                  \
        DISPATCH_PATTERN(half, NRanks);                                                                                \
    }                                                                                                                  \
    else if (params.dtype == nvinfer1::DataType::kBF16)                                                                \
    {                                                                                                                  \
        DISPATCH_PATTERN(__hip_bfloat16, NRanks);                                                                       \
    }                                                                                                                  \
    else if (params.dtype == nvinfer1::DataType::kFLOAT)                                                               \
    {                                                                                                                  \
        DISPATCH_PATTERN(float, NRanks);                                                                               \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        TLLM_CHECK_WITH_INFO(false, "allreduce_fusion_kernel: unsupported dtype!");                                    \
    }

#define DISPATCH_RANKS(NRanks)                                                                                         \
    if (params.nranks == NRanks)                                                                                       \
    {                                                                                                                  \
        DISPATCH_DTYPE(NRanks);                                                                                        \
    }

    TLLM_CHECK(params.allreduce_in && params.residual_in && params.rms_gamma);
    TLLM_CHECK(params.size % params.hidden_dim == 0);
    bool fp32_acc = use_fp32_acc();
    DISPATCH_RANKS(2);
    DISPATCH_RANKS(4);
    DISPATCH_RANKS(8);
    DISPATCH_RANKS(16);
    TLLM_CHECK_WITH_INFO(false, "allreduce_fusion_kernel: unsupported ranks number!");
}
}; // namespace tensorrt_llm::kernels::ar_fusion
