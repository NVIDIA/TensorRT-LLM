#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "mnnvlAllreduceKernels.h"
#include <hip/hip_cooperative_groups.h>
#include <cstddef>
#include <cstdint>
#include <cuda/atomic>
#include <hip/hip_bf16.h>
#include <cuda_pipeline.h>
#include <tuple>
#include <type_traits>

#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/dataType.h"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/common/lamportUtils.cuh"
#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"

namespace tensorrt_llm::kernels::mnnvl
{

using tensorrt_llm::common::isNegZero;
using tensorrt_llm::common::LamportFlags;
using tensorrt_llm::common::cuda_cast;
using tensorrt_llm::common::getMultiProcessorCount;
using tensorrt_llm::common::getDTypeSize;

// Guard the helper function used for this kernel.
namespace detail
{
template <typename PackedType, typename T>
union PackedVec
{
    PackedType packed;
    T elements[sizeof(PackedType) / sizeof(T)];

    __device__ PackedVec& operator+=(PackedVec& other)
    {
#pragma unroll
        for (int i = 0; i < sizeof(PackedType) / sizeof(T); i++)
        {
            elements[i] += other.elements[i];
        }
        return *this;
    }

    __device__ PackedVec operator+(PackedVec& other)
    {
        PackedVec result;
#pragma unroll
        for (int i = 0; i < sizeof(PackedType) / sizeof(T); i++)
        {
            result.elements[i] = elements[i] + other.elements[i];
        }
        return result;
    }
};

template <typename PackedType, typename T>
inline __device__ PackedType loadPacked(T* ptr)
{
    return *reinterpret_cast<PackedType*>(ptr);
}

template <typename PackedType, typename T>
inline __device__ const PackedType loadPacked(T const* ptr)
{
    return *reinterpret_cast<PackedType const*>(ptr);
}

template <typename PackedType>
inline __device__ PackedType loadPackedVolatile(void const* ptr)
{
    static_assert(sizeof(PackedType) == 0, "Not implemented");
    return PackedType{};
}

template <>
inline __device__ float4 loadPackedVolatile<float4>(void const* ptr)
{
    float4 returnValue;
    asm volatile("ld.volatile.global.v4.f32 {%0, %1, %2, %3}, [%4];\n"
                 : "=f"(returnValue.x), "=f"(returnValue.y), "=f"(returnValue.z), "=f"(returnValue.w)
                 : "l"(ptr));
    return returnValue;
}

template <>
inline __device__ float2 loadPackedVolatile<float2>(void const* ptr)
{
    float2 returnValue;
    asm volatile("ld.volatile.global.v2.f32 {%0, %1}, [%2];\n" : "=f"(returnValue.x), "=f"(returnValue.y) : "l"(ptr));
    return returnValue;
}

template <typename T_IN>
inline __device__ void copyF4(T_IN* dst, T_IN const* src)
{
    float4* dst4 = reinterpret_cast<float4*>(dst);
    float4 const* src4 = reinterpret_cast<float4 const*>(src);
    __pipeline_memcpy_async(dst4, src4, sizeof(float4));
}

uint32_t constexpr kWARP_SIZE = 32U;
uint32_t constexpr kLOG2_WARP_SIZE = 5U;
uint32_t constexpr kLANE_ID_MASK = 0x1f;

template <typename T>
inline __device__ T warpReduceSumPartial(T val)
{
    int laneId = threadIdx.x & kLANE_ID_MASK;
    // We make sure only the last warp will call this function
    int warpSize = blockDim.x - (threadIdx.x & ~(kWARP_SIZE - 1));
    unsigned int active_mask = (1U << warpSize) - 1;

#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1)
    {
        int targetLane = laneId ^ mask;
        auto tmp = __shfl_xor_sync(active_mask, val, mask, kWARP_SIZE);
        val += targetLane < warpSize ? tmp : 0;
    }
    return val;
}

// SYNC:
//  - True: share the sume across all threads
//  - False: only thread 0 get the sum; Other thread's value is undefined.
template <typename T, bool SYNC = false>
inline __device__ T blockReduceSumPartial(T val)
{
    __shared__ T smem[kWARP_SIZE + 1];
    int laneId = threadIdx.x & kLANE_ID_MASK;
    int warpId = threadIdx.x >> kLOG2_WARP_SIZE;
    int warpNum = (blockDim.x + kWARP_SIZE - 1) >> kLOG2_WARP_SIZE; // Ceiling division to include partial warps

    val = (warpId == warpNum - 1) ? warpReduceSumPartial(val) : tensorrt_llm::common::warpReduceSum(val);
    if (laneId == 0)
    {
        smem[warpId] = val;
    }
    __syncthreads();

    if (warpId == 0)
    {
        val = (laneId < warpNum) ? smem[laneId] : (T) 0.f;
        // Need to consider the corner case where we only have one warp and it is partial
        val = (warpNum == 1) ? warpReduceSumPartial(val) : tensorrt_llm::common::warpReduceSum(val);

        if constexpr (SYNC)
        {
            if (laneId == 0)
            {
                smem[warpId] = val;
            }
        }
    }
    if constexpr (SYNC)
    {
        __syncthreads();
        val = smem[0];
    }
    return val;
}

template <typename T, bool SYNC = false>
inline __device__ T blockReduceSum(T val)
{
    bool hasPartialWarp = (blockDim.x & kLANE_ID_MASK) != 0;
    if (hasPartialWarp)
    {
        return blockReduceSumPartial<T, SYNC>(val);
    }
    else
    {
        return tensorrt_llm::common::blockReduceSum<T>(val);
    }
}

// We have to define this again since the one in mathUtils.h is shadowed by the one from cudaUtils.h, which is a
// host-only function!
template <typename T>
inline __device__ __host__ T divUp(T m, T n)
{
    return (m + n - 1) / n;
}

// A helper function to tune the grid configuration for fused oneshot and rmsnorm kernels
// Return (block_size, cluster_size, loads_per_thread)
std::tuple<int, int, int> adjustGridConfig(int numTokens, int dim, int eltsPerThread)
{
    // Start with preferred block_size and cluster_size
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    int clusterSize = 8;
#else
    int clusterSize = 1;
#endif
    int blockSize = 128;
    // ========================== Adjust the grid configuration ==========================
    int threadsNeeded = divUp(dim, eltsPerThread);
    int loadsPerThread = 1;

    blockSize = divUp(threadsNeeded, clusterSize);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    while (threadsNeeded % clusterSize != 0 && clusterSize > 1)
    {
        clusterSize /= 2;
    }
    blockSize = divUp(threadsNeeded, clusterSize);
    while (blockSize < 128 && clusterSize >= 2)
    {
        blockSize *= 2;
        clusterSize /= 2;
    }
    int smCount = getMultiProcessorCount();
    while (numTokens * clusterSize > smCount && clusterSize > 1 && blockSize <= 512)
    {
        blockSize *= 2;
        clusterSize /= 2;
    }
#endif

    // Trying to scale up use multiple loads or CGA
    while (blockSize > 1024)
    {
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        if (clusterSize < 8)
        {
            clusterSize = clusterSize << 1;
        }
        else
        {
            break;
        }
#else
        if (loadsPerThread < 8)
        {
            loadsPerThread += 1;
        }
        else
        {
            break;
        }
#endif
        blockSize = divUp(threadsNeeded, clusterSize * loadsPerThread);
    }
    return {blockSize, clusterSize, loadsPerThread};
}

} // namespace detail

using detail::PackedVec;
using detail::loadPacked;
using detail::loadPackedVolatile;
using detail::blockReduceSum;
using detail::divUp;
using detail::copyF4;

template <uint8_t WorldSize, typename T, bool RMSNormFusion = false, typename PackedType = float4>
__global__ void __launch_bounds__(1024) oneshotAllreduceFusionKernel(T* outputPtr, T* prenormedPtr, T const* shardPtr,
    T const* residualInPtr, T const* gammaPtr, T** inputPtrs, T* mcastPtr, int const numTokens, int const tokenDim,
    float epsilon, int const rank, uint32_t* bufferFlags)
{
    constexpr int kELTS_PER_THREAD = sizeof(PackedType) / sizeof(T);
    constexpr int kLAMPORT_ELTS_PER_PACKED = sizeof(PackedType) / sizeof(float);
    constexpr uint32_t kELT_SIZE = sizeof(T);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    namespace cg = cooperative_groups;
    cg::cluster_group cluster = cg::this_cluster();
    int packedIdx = cluster.thread_rank();
    int token = blockIdx.x;
    int threadOffset = token * tokenDim + packedIdx * kELTS_PER_THREAD;

    cudaGridDependencySynchronize();
#else
    int packedIdx = blockIdx.y * blockDim.x + threadIdx.x;
    int token = blockIdx.x;
    // Offset w.r.t. the input shard
    int threadOffset = token * tokenDim + packedIdx * kELTS_PER_THREAD;
#endif

    // We only use 1 stage for the oneshot allreduce
    LamportFlags<PackedType> flag(bufferFlags, 1);
    T* stagePtrMcast = reinterpret_cast<T*>(flag.getCurLamportBuf(mcastPtr, 0));
    T* stagePtrLocal = reinterpret_cast<T*>(flag.getCurLamportBuf(inputPtrs[rank], 0));

    if (packedIdx * kELTS_PER_THREAD >= tokenDim)
    {
        flag.clearDirtyLamportBuf(inputPtrs[rank], -1);
        return;
    }

    // ==================== Broadcast tokens to each rank =============================
    PackedVec<PackedType, T> val;
    val.packed = loadPacked<PackedType>(&shardPtr[threadOffset]);
#pragma unroll
    for (int i = 0; i < kELTS_PER_THREAD; i++)
    {
        if (isNegZero(val.elements[i]))
            val.elements[i] = cuda_cast<T, float>(0.f);
    }

    reinterpret_cast<PackedType*>(&stagePtrMcast[token * tokenDim * WorldSize + rank * tokenDim])[packedIdx]
        = val.packed;

    flag.ctaArrive();
    // ======================= Lamport Sync and clear the output buffer from previous iteration
    // =============================
    flag.clearDirtyLamportBuf(inputPtrs[rank], -1);

    PackedVec<PackedType, float> valuesLamport[WorldSize];
    while (1)
    {
        bool valid = true;
#pragma unroll
        for (int r = 0; r < WorldSize; r++)
        {
            valuesLamport[r].packed = loadPackedVolatile<PackedType>(
                &stagePtrLocal[token * tokenDim * WorldSize + r * tokenDim + packedIdx * kELTS_PER_THREAD]);

#pragma unroll
            for (int i = 0; i < kLAMPORT_ELTS_PER_PACKED; i++)
            {
                valid &= !isNegZero(valuesLamport[r].elements[i]);
            }
        }
        if (valid)
        {
            break;
        }
    }

    auto values = reinterpret_cast<PackedVec<PackedType, T>*>(valuesLamport);
    // ======================= Reduction =============================
    float accum[kELTS_PER_THREAD];
    PackedVec<PackedType, T> packedAccum;

#pragma unroll
    for (int i = 0; i < kELTS_PER_THREAD; i++)
    {
        accum[i] = cuda_cast<float, T>(values[0].elements[i]);
    }

#pragma unroll
    for (int r = 1; r < WorldSize; r++)
    {
#pragma unroll
        for (int i = 0; i < kELTS_PER_THREAD; i++)
        {
            accum[i] += cuda_cast<float, T>(values[r].elements[i]);
        }
    }

#pragma unroll
    for (int i = 0; i < kELTS_PER_THREAD; i++)
    {
        packedAccum.elements[i] = cuda_cast<T, float>(accum[i]);
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
    if constexpr (RMSNormFusion)
    {
        // =============================== Residual ===============================
        PackedVec<PackedType, T> residualIn;
        residualIn.packed = *reinterpret_cast<PackedType const*>(&residualInPtr[threadOffset]);
        packedAccum += residualIn;
        *reinterpret_cast<PackedType*>(&prenormedPtr[threadOffset]) = packedAccum.packed;
        // =============================== Rmsnorm ================================
        PackedVec<PackedType, T> gamma;
        gamma.packed = *reinterpret_cast<PackedType const*>(&gammaPtr[packedIdx * kELTS_PER_THREAD]);

        float threadSum = 0.F;
        __shared__ float sharedVal; // Temporary variable to share the sum within block
#pragma unroll
        for (int i = 0; i < kELTS_PER_THREAD; i++)
        {
            // FIXME: Use float square if accuracy issue
            threadSum += cuda_cast<float, T>(packedAccum.elements[i] * packedAccum.elements[i]);
        }
        float tokenSum = blockReduceSum<float, false>(threadSum);
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        namespace cg = cooperative_groups;
        cg::cluster_group cluster = cg::this_cluster();
        if (cluster.num_blocks() > 1)
        {
            // Need to reduce over the entire cluster
            if (threadIdx.x == 0)
            {
                sharedVal = tokenSum;
                tokenSum = 0.F;
            }
            cluster.sync();
            if (threadIdx.x == 0)
            {
                for (int i = 0; i < cluster.num_blocks(); ++i)
                {
                    tokenSum += *cluster.map_shared_rank(&sharedVal, i);
                }
            }
            cluster.sync();
        }
#endif
        if (threadIdx.x == 0)
        {
            sharedVal = rsqrtf(tokenSum / tokenDim + epsilon);
        }
        __syncthreads();
#pragma unroll
        for (int i = 0; i < kELTS_PER_THREAD; i++)
        {
            packedAccum.elements[i] = cuda_cast<T, float>(
                cuda_cast<float, T>(packedAccum.elements[i]) * sharedVal * cuda_cast<float, T>(gamma.elements[i]));
        }
    }
    reinterpret_cast<PackedType*>(&outputPtr[threadOffset])[0] = packedAccum.packed;
    flag.waitAndUpdate({static_cast<uint32_t>(numTokens * tokenDim * WorldSize * kELT_SIZE), 0, 0, 0});
}

using detail::adjustGridConfig;

void oneshotAllreduceFusionOp(AllReduceFusionParams const& params)
{
    int const numTokens = params.numTokens;
    int const tokenDim = params.tokenDim;
    int const eltsPerThread = sizeof(float4) / getDTypeSize(params.dType);

    auto [blockSize, clusterSize, loadsPerThread] = adjustGridConfig(numTokens, tokenDim, eltsPerThread);
    dim3 grid(numTokens, clusterSize, 1);

    TLLM_CHECK_WITH_INFO(blockSize <= 1024 && loadsPerThread == 1,
        "Hidden Dimension %d exceeds the maximum supported hidden dimension (%d)", tokenDim,
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        1024 * 8 * eltsPerThread);
#else
        1024 * eltsPerThread);
#endif

    TLLM_LOG_DEBUG(
        "[MNNVL AllReduceOneShot] Dispatch: grid size: (%d, %d, 1), block_size: %d, cluster_size: %d, "
        "loads_per_thread: %d, "
        "threads_needed: %d",
        numTokens, clusterSize, blockSize, clusterSize, loadsPerThread, divUp(tokenDim, eltsPerThread));

    hipLaunchAttribute attrs[2];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL() ? 1 : 0;
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    attrs[1].id = cudaLaunchAttributeClusterDimension;
    attrs[1].val.clusterDim.x = 1;
    attrs[1].val.clusterDim.y = clusterSize;
    attrs[1].val.clusterDim.z = 1;
#endif

    hipLaunchConfig_t config
    {
        .gridDim = grid, .blockDim = blockSize, .dynamicSmemBytes = 0, .stream = params.stream, .attrs = attrs,
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
        .numAttrs = 2,
#else
        .numAttrs = 1,
#endif
    };

#define LAUNCH_ALLREDUCE_KERNEL(WORLD_SIZE, T, RMSNORM)                                                                \
    TLLM_CUDA_CHECK(cudaLaunchKernelEx(&config, &oneshotAllreduceFusionKernel<WORLD_SIZE, T, RMSNORM>, output,         \
        residualOut, input, residualIn, gamma, ucPtrs, mcPtr, numTokens, tokenDim, static_cast<float>(params.epsilon), \
        params.rank, params.bufferFlags));
#define DISPATCH_ALLREDUCE_KERNEL(WORLD_SIZE, T)                                                                       \
    if (params.rmsNormFusion)                                                                                          \
    {                                                                                                                  \
        LAUNCH_ALLREDUCE_KERNEL(WORLD_SIZE, T, true);                                                                  \
    }                                                                                                                  \
    else                                                                                                               \
    {                                                                                                                  \
        LAUNCH_ALLREDUCE_KERNEL(WORLD_SIZE, T, false);                                                                 \
    }
    // C++17 compatible alternative using a template function
    auto dispatchImpl = [&](auto* type_ptr) -> bool
    {
        using T = std::remove_pointer_t<decltype(type_ptr)>;
        T** ucPtrs = reinterpret_cast<T**>(params.bufferPtrsDev);
        T* mcPtr = reinterpret_cast<T*>(params.multicastPtr);
        T* output = reinterpret_cast<T*>(params.output);
        T* residualOut = reinterpret_cast<T*>(params.residualOut);
        T const* input = reinterpret_cast<T const*>(params.input);
        T const* residualIn = reinterpret_cast<T const*>(params.residualIn);
        T const* gamma = reinterpret_cast<T const*>(params.gamma);

        switch (params.nRanks)
        {
            // FIXME: Do we need other world sizes?
        case 2: DISPATCH_ALLREDUCE_KERNEL(2, T); return true;
        case 4: DISPATCH_ALLREDUCE_KERNEL(4, T); return true;
        case 8: DISPATCH_ALLREDUCE_KERNEL(8, T); return true;
        case 16: DISPATCH_ALLREDUCE_KERNEL(16, T); return true;
        case 32: DISPATCH_ALLREDUCE_KERNEL(32, T); return true;
        case 64: DISPATCH_ALLREDUCE_KERNEL(64, T); return true;
        }
        return false;
    };
#undef LAUNCH_ALLREDUCE_KERNEL
#undef DISPATCH_ALLREDUCE_KERNEL
    bool launched = (params.dType == nvinfer1::DataType::kBF16 && dispatchImpl((__hip_bfloat16*) nullptr))
        || (params.dType == nvinfer1::DataType::kFLOAT && dispatchImpl((float*) nullptr))
        || (params.dType == nvinfer1::DataType::kHALF && dispatchImpl((__nv_half*) nullptr));
    if (!launched)
    {
        TLLM_CHECK_WITH_INFO(false, "Failed to dispatch MNNVL AllReduceOneShot kernel.");
    }
}

enum MNNVLTwoShotStage : uint8_t
{
    SCATTER = 0,
    BROADCAST = 1,
    NUM_STAGES = 2,
};

template <uint8_t WorldSize, typename T, typename PackedType = float4>
__global__ __launch_bounds__(128) void twoshotAllreduceKernel(T* outputPtr, T const* shardPtr, T** inputPtrs,
    T* mcastPtr, uint32_t const numTokens, uint32_t const tokenDim, uint32_t const rank, uint32_t* bufferFlags,
    bool const wait_for_results)
{
    constexpr int kELTS_PER_THREAD = sizeof(PackedType) / sizeof(T);
    constexpr int kLAMPORT_ELTS_PER_PACKED = sizeof(PackedType) / sizeof(float);
    constexpr uint32_t kELT_SIZE = sizeof(T);

    int packedIdx = blockIdx.y * blockDim.x + threadIdx.x;
    int token = blockIdx.x;
    // Offset w.r.t. the input shard
    int threadOffset = token * tokenDim + packedIdx * kELTS_PER_THREAD;

    int destRank = token % WorldSize;
    int destTokenOffset = token / WorldSize;
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaGridDependencySynchronize();
#endif
    LamportFlags<PackedType> flag(bufferFlags, MNNVLTwoShotStage::NUM_STAGES);

    T* scatterBufLocal = reinterpret_cast<T*>(flag.getCurLamportBuf(inputPtrs[rank], MNNVLTwoShotStage::SCATTER));
    T* scatterBufDest = reinterpret_cast<T*>(flag.getCurLamportBuf(inputPtrs[destRank], MNNVLTwoShotStage::SCATTER));
    T* broadcastBufW = reinterpret_cast<T*>(flag.getCurLamportBuf(mcastPtr, MNNVLTwoShotStage::BROADCAST));
    T* broadcastBufR = reinterpret_cast<T*>(flag.getCurLamportBuf(inputPtrs[rank], MNNVLTwoShotStage::BROADCAST));

    // Make sure the clear function is called before OOB thread exits
    if (packedIdx * kELTS_PER_THREAD >= tokenDim)
    {
        flag.clearDirtyLamportBuf(inputPtrs[rank], -1);
        return;
    }

    // =============================== Scatter ===============================

    // Load vectorized data
    PackedVec<PackedType, T> val;
    val.packed = loadPacked<PackedType>(&shardPtr[threadOffset]);
#pragma unroll
    for (int i = 0; i < kELTS_PER_THREAD; i++)
    {
        if (isNegZero(val.elements[i]))
        {
            val.elements[i] = cuda_cast<T, float>(0.F);
        }
    }

    // Store vectorized data
    reinterpret_cast<PackedType*>(&scatterBufDest[destTokenOffset * tokenDim * WorldSize + rank * tokenDim])[packedIdx]
        = val.packed;

    flag.clearDirtyLamportBuf(inputPtrs[rank], MNNVLTwoShotStage::SCATTER);

    // =============================== Reduction and Broadcast ===============================

    if ((token % WorldSize) == rank)
    {
        int localToken = token / WorldSize;
        float accum[kELTS_PER_THREAD] = {0.F};

        // Use float as we only check each float value for validity
        PackedVec<PackedType, float> valuesLamport[WorldSize];
        while (1)
        {
            bool valid = true;
#pragma unroll
            for (int r = 0; r < WorldSize; r++)
            {
                valuesLamport[r].packed = loadPackedVolatile<PackedType>(
                    &scatterBufLocal[localToken * tokenDim * WorldSize + r * tokenDim + packedIdx * kELTS_PER_THREAD]);

                // Check validity across all elements
#pragma unroll
                for (int i = 0; i < kLAMPORT_ELTS_PER_PACKED; i++)
                {
                    valid &= !isNegZero(valuesLamport[r].elements[i]);
                }
            }
            if (valid)
            {
                break;
            }
        }

        // Now we view it as the value for reduction
        auto values = reinterpret_cast<PackedVec<PackedType, T>*>(valuesLamport);
#pragma unroll
        for (int r = 0; r < WorldSize; r++)
        {

#pragma unroll
            for (int i = 0; i < kELTS_PER_THREAD; i++)
            {
                accum[i] += cuda_cast<float, T>(values[r].elements[i]);
            }
        }

        // Store vectorized result
        PackedVec<PackedType, T> packedAccum;
#pragma unroll
        for (int i = 0; i < kELTS_PER_THREAD; i++)
        {
            packedAccum.elements[i] = cuda_cast<T, float>(accum[i]);
        }
        reinterpret_cast<PackedType*>(&broadcastBufW[token * tokenDim])[packedIdx] = packedAccum.packed;
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
    flag.clearDirtyLamportBuf(inputPtrs[rank], MNNVLTwoShotStage::BROADCAST);

    // Optionally wait for results if the next layer isn't doing the Lamport check
    if (wait_for_results)
    {
        // Update the atomic counter to indicate the block has read the offsets
        flag.ctaArrive();

        PackedVec<PackedType, float> valLamport;
        valLamport.packed = loadPackedVolatile<PackedType>(&broadcastBufR[threadOffset]);
        while (isNegZero(valLamport.elements[0]))
        {
            valLamport.packed = loadPackedVolatile<PackedType>(&broadcastBufR[threadOffset]);
        }
        if (outputPtr)
        {
            reinterpret_cast<PackedType*>(&outputPtr[threadOffset])[0] = valLamport.packed;
        }

        // Update the buffer flags
        flag.waitAndUpdate({static_cast<uint32_t>(divUp<uint32_t>(numTokens, WorldSize) * WorldSize * tokenDim
                                * kELT_SIZE),                        // Clear Size for scatter stage
            static_cast<uint32_t>(numTokens * tokenDim * kELT_SIZE), // Clear Size for broadcast stage
            0, 0});
        // If not wait for results, we will rely on the following kernel to update the buffer
    }
}

// This kernel works performant when loads_per_thread is 1.
// For this mode, we are able to support up to 1024 (threads) x 8 (elements) = 8192 hidden dimension.
// There are two options for further scaling up:
//      1. Use CGA if supported. It expands the hidden dimension to 8k x 8 = 64k.
//      2. Set loads_per_thread >1. Which can be used if CGA is not supported. Note that this will be limited by the
//      shared memory size and register count.
template <typename T_IN, typename T_OUT, int LoadsPerThread = 1>
__global__ __launch_bounds__(1024) void rmsNormLamport(T_IN* outputPreNorm, T_OUT* outputNorm, T_IN* bufferInput,
    T_IN const* gamma, float epsilon, T_IN const* residual, uint32_t numTokens, uint32_t dim, uint32_t worldSize,
    uint32_t* bufferFlags)
{
    static_assert(std::is_same_v<T_IN, T_OUT>, "T_IN and T_OUT must be the same type");
    static int const kELTS_PER_LOAD = sizeof(float4) / sizeof(T_IN);

    uint32_t const token = blockIdx.x;
    uint32_t const blockSize = blockDim.x;
    uint32_t const threadOffset = threadIdx.x;

    uint32_t numThreads = blockSize;
    uint32_t clusterSize = 1;
    uint32_t blockOffset = 0;
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    namespace cg = cooperative_groups;
    cg::cluster_group cluster = cg::this_cluster();
    numThreads = cluster.num_threads();
    clusterSize = cluster.num_blocks();
    blockOffset = cluster.block_rank();
#endif
    uint32_t const dimPadded = divUp(dim, kELTS_PER_LOAD * numThreads) * kELTS_PER_LOAD * numThreads;
    uint32_t const elemsPerThread = dimPadded / numThreads;
    uint32_t const loadStride = blockSize;

    extern __shared__ uint8_t smem[];
    float rInput[LoadsPerThread * kELTS_PER_LOAD];
    uint32_t offsets[LoadsPerThread * kELTS_PER_LOAD];

    uint32_t const smemBufferSize = blockSize * elemsPerThread * sizeof(T_IN);
    T_IN* smemInput = (T_IN*) &smem[0];
    T_IN* smemResidual = (T_IN*) &smem[smemBufferSize];
    T_IN* smemGamma = (T_IN*) &smem[2 * smemBufferSize];

    LamportFlags<float4> flag(bufferFlags, MNNVLTwoShotStage::NUM_STAGES);
    T_IN* input = reinterpret_cast<T_IN*>(
        flag.getCurLamportBuf(reinterpret_cast<void*>(bufferInput), MNNVLTwoShotStage::BROADCAST));

#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    cudaTriggerProgrammaticLaunchCompletion();
#endif
    // The offset that current thread should load from. Note that the hidden dimension is split by CGA size and each
    // block loads a contiguous chunk;
    // The size of chunk that each block processes
    uint32_t const blockChunkSize = divUp(dim, clusterSize * kELTS_PER_LOAD) * kELTS_PER_LOAD;
    uint32_t const blockLoadOffset = token * dim + blockOffset * blockChunkSize;

#pragma unroll
    for (uint32_t i = 0; i < LoadsPerThread; i++)
    {
        // Each block load a contiguous chunk of tokens
        uint32_t const threadLoadOffset = (i * loadStride + threadOffset) * kELTS_PER_LOAD;
        offsets[i] = blockLoadOffset + threadLoadOffset;
    }

#pragma unroll
    for (uint32_t i = 0; i < LoadsPerThread; i++)
    {
        uint32_t const threadLoadOffset = (i * loadStride + threadOffset) * kELTS_PER_LOAD;
        if (blockOffset * blockChunkSize + threadLoadOffset < dim)
        {
            copyF4(&smemResidual[threadLoadOffset], &residual[blockLoadOffset + threadLoadOffset]);
        }
    }
    __pipeline_commit();
#pragma unroll
    for (uint32_t i = 0; i < LoadsPerThread; i++)
    {
        uint32_t const threadLoadOffset = (i * loadStride + threadOffset) * kELTS_PER_LOAD;
        if (blockOffset * blockChunkSize + threadLoadOffset < dim)
        {
            copyF4(&smemGamma[threadLoadOffset], &gamma[blockOffset * blockChunkSize + threadLoadOffset]);
        }
    }
    __pipeline_commit();

    flag.ctaArrive();
    bool valid = false;
    // ACQBLK if not lamport
    while (!valid)
    {
        valid = true;
#pragma unroll
        for (uint32_t i = 0; i < LoadsPerThread; i++)
        {
            uint32_t threadLoadOffset = (i * loadStride + threadOffset) * kELTS_PER_LOAD;

            if (blockOffset * blockChunkSize + threadLoadOffset < dim)
            {

                float4* dst4 = reinterpret_cast<float4*>(&smemInput[threadLoadOffset]);
                float4 const* src4 = reinterpret_cast<float4 const*>(&input[offsets[i]]);

                float4 value = loadPackedVolatile<float4>(src4);
                // Assume that the 16B were written atomically, so we only need to check one value
                valid &= !isNegZero(value.x);
                *dst4 = value;
            }
        }
    }

    __pipeline_wait_prior(1);
    __syncthreads();

    float threadSum = 0.f;
#pragma unroll
    for (int i = 0; i < LoadsPerThread; i++)
    {
        int threadLoadOffset = (i * loadStride + threadOffset) * kELTS_PER_LOAD;
        if (blockOffset * blockChunkSize + threadLoadOffset < dim)
        {
            PackedVec<float4, T_IN> inp{.packed = loadPacked<float4>(&smemInput[threadLoadOffset])};
            PackedVec<float4, T_IN> res{.packed = loadPacked<float4>(&smemResidual[threadLoadOffset])};

            PackedVec<float4, T_IN> inp_plus_res = inp + res;
#pragma unroll
            for (int j = 0; j < kELTS_PER_LOAD; j++)
            {
                rInput[i * kELTS_PER_LOAD + j] = cuda_cast<float, T_IN>(inp_plus_res.elements[j]);
                threadSum += cuda_cast<float, T_IN>(inp_plus_res.elements[j] * inp_plus_res.elements[j]);
            }

            *reinterpret_cast<float4*>(&outputPreNorm[blockLoadOffset + threadLoadOffset]) = inp_plus_res.packed;
        }
    }

    __pipeline_wait_prior(0);

    // Sum is only used in thread 0!
    float clusterSum = blockReduceSum<float, false>(threadSum);

    float rcpRms;
    __shared__ float sharedVal;
    // Use CGA Reduction if supported
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    if (cluster.num_blocks() > 1)
    {
        // Need to reduce over the entire cluster
        if (threadIdx.x == 0)
        {
            sharedVal = clusterSum;
            clusterSum = 0.F;
        }
        cluster.sync();
        if (threadIdx.x == 0)
        {
            for (int i = 0; i < clusterSize; ++i)
            {
                clusterSum += *cluster.map_shared_rank(&sharedVal, i);
            }
        }
        cluster.sync();
    }
#endif

    if (threadIdx.x == 0)
    {
        sharedVal = rsqrtf(clusterSum / dim + epsilon);
    }
    __syncthreads();
    rcpRms = sharedVal;

#pragma unroll
    for (int i = 0; i < LoadsPerThread; i++)
    {
        PackedVec<float4, T_OUT> r_out;
        uint32_t threadLoadOffset = (i * loadStride + threadOffset) * kELTS_PER_LOAD;
        if (blockOffset * blockChunkSize + threadLoadOffset < dim)
        {
            PackedVec<float4, T_IN> gamma = {.packed = loadPacked<float4>(&smemGamma[threadLoadOffset])};

#pragma unroll
            for (uint32_t j = 0; j < kELTS_PER_LOAD; j++)
            {
                r_out.elements[j]
                    = cuda_cast<T_OUT, float>(cuda_cast<float, T_IN>(gamma.elements[j]) * rInput[j] * rcpRms);
            }

            *reinterpret_cast<float4*>(&outputNorm[blockLoadOffset + threadLoadOffset]) = r_out.packed;
        }
    }
    constexpr int kELTS_SIZE = sizeof(T_IN);

    // Update the buffer pointers
    flag.waitAndUpdate({static_cast<uint32_t>(divUp<uint32_t>(numTokens, worldSize) * worldSize * dim * kELTS_SIZE),
        static_cast<uint32_t>(numTokens * dim * kELTS_SIZE), 0, 0});
}

void twoshotAllreduceFusionOp(AllReduceFusionParams const& params)
{
    int const numTokens = params.numTokens;
    int const tokenDim = params.tokenDim;
    int const numEltsPerThread = sizeof(float4) / getDTypeSize(params.dType);
    TLLM_CHECK_WITH_INFO(tokenDim % numEltsPerThread == 0, "[MNNVL AllReduceTwoShot] token_dim must be divisible by %d",
        numEltsPerThread);

    int const arNumThreads = divUp(tokenDim, numEltsPerThread);
    int const arNumBlocksPerToken = divUp(arNumThreads, 128);

    dim3 arGrid(numTokens, arNumBlocksPerToken);

    hipLaunchAttribute arAttrs[1];
    arAttrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    arAttrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL() ? 1 : 0;

    hipLaunchConfig_t arConfig{
        .gridDim = arGrid,
        .blockDim = 128,
        .dynamicSmemBytes = 0,
        .stream = params.stream,
        .attrs = arAttrs,
        .numAttrs = 1,
    };

    TLLM_LOG_DEBUG(
        "[MNNVL AllReduceTwoShot] Dispatch: grid size: (%d, %d, 1), block_size: 128", numTokens, arNumBlocksPerToken);

#define LAUNCH_ALLREDUCE_KERNEL(WORLD_SIZE, T)                                                                         \
    TLLM_CUDA_CHECK(cudaLaunchKernelEx(&arConfig, &twoshotAllreduceKernel<WORLD_SIZE, T>, output, input, ucPtrs,       \
        mcastPtr, numTokens, tokenDim, params.rank, params.bufferFlags, (!params.rmsNormFusion)));
    auto dispatchAR = [&](auto* type_ptr) -> bool
    {
        using T = std::remove_pointer_t<decltype(type_ptr)>;
        T** ucPtrs = reinterpret_cast<T**>(params.bufferPtrsDev);
        T* mcastPtr = reinterpret_cast<T*>(params.multicastPtr);
        T* output = reinterpret_cast<T*>(params.output);
        T const* input = reinterpret_cast<T const*>(params.input);
        switch (params.nRanks)
        {
        case 2: LAUNCH_ALLREDUCE_KERNEL(2, T); return true;
        case 4: LAUNCH_ALLREDUCE_KERNEL(4, T); return true;
        case 8: LAUNCH_ALLREDUCE_KERNEL(8, T); return true;
        case 16: LAUNCH_ALLREDUCE_KERNEL(16, T); return true;
        case 32: LAUNCH_ALLREDUCE_KERNEL(32, T); return true;
        case 64: LAUNCH_ALLREDUCE_KERNEL(64, T); return true;
        }
        return false;
    };

#undef LAUNCH_ALLREDUCE_KERNEL

    bool launched = (params.dType == nvinfer1::DataType::kFLOAT && dispatchAR((float*) nullptr))
        || (params.dType == nvinfer1::DataType::kBF16 && dispatchAR((__hip_bfloat16*) nullptr))
        || (params.dType == nvinfer1::DataType::kHALF && dispatchAR((__nv_half*) nullptr));
    if (!launched)
    {
        TLLM_CHECK_WITH_INFO(false, "[MNNVL AllReduceTwoShot] Failed to dispatch twoshotAllreduce kernel.");
    }
    // Launch the rmsnorm lamport kernel if fusion is enabled
    if (params.rmsNormFusion)
    {
        auto gridConfig = adjustGridConfig(numTokens, tokenDim, numEltsPerThread);
        int rnBlockSize = std::get<0>(gridConfig);
        int rnClusterSize = std::get<1>(gridConfig);
        int rnLoadsPerThread = std::get<2>(gridConfig);

        int rnNumThreads = rnClusterSize * rnBlockSize;
        dim3 rnGrid(numTokens, rnClusterSize, 1);
        hipLaunchConfig_t rnConfig;
        hipLaunchAttribute rnAttrs[2];
        rnConfig.stream = params.stream;
        rnConfig.gridDim = rnGrid;
        rnConfig.blockDim = rnBlockSize;
        rnConfig.attrs = rnAttrs;
        rnAttrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
        rnAttrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL() ? 1 : 0;
#ifndef DISABLE_CGA
        rnAttrs[1].id = cudaLaunchAttributeClusterDimension;
        rnAttrs[1].val.clusterDim.x = 1;
        rnAttrs[1].val.clusterDim.y = rnClusterSize;
        rnAttrs[1].val.clusterDim.z = 1;
        rnConfig.numAttrs = 2;
#else
        rnConfig.numAttrs = 1;
#endif

        bool const rnUseCGA = rnClusterSize > 1;
        int const dimPadded = divUp(tokenDim, numEltsPerThread * rnNumThreads) * numEltsPerThread * rnNumThreads;
        int const iters = dimPadded / rnNumThreads;

        size_t const smemSize = 3 * rnBlockSize * iters * getDTypeSize(params.dType);

        TLLM_LOG_DEBUG(
            "[MNNVL AllReduceTwoShotRMSNorm] Dispatch: grid size: (%d, %d, 1), block_size: %d, cluster_size: %d, "
            "loads_per_thread: %d, "
            "threads_needed: %d",
            numTokens, rnClusterSize, rnBlockSize, rnClusterSize, rnLoadsPerThread, divUp(tokenDim, numEltsPerThread));

#define RUN_RMSNORM_KERNEL(T_IN, T_OUT, LOADS_PER_THREAD)                                                              \
    TLLM_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(                                                                              \
        &rmsNormLamport<T_IN), T_OUT, LOADS_PER_THREAD>, hipFuncAttributeMaxDynamicSharedMemorySize, smemSize));       \
    rnConfig.dynamicSmemBytes = smemSize;                                                                              \
    TLLM_CUDA_CHECK(cudaLaunchKernelEx(&rnConfig, &rmsNormLamport<T_IN, T_OUT, LOADS_PER_THREAD>, residualOut, output, \
        bufferInput, gamma, static_cast<float>(params.epsilon), residualIn, numTokens, tokenDim, params.nRanks,        \
        params.bufferFlags));

        // C++ 17 does not support capturing structured bindings
        auto dispatchRN = [&, rnLoadsPerThread](auto* type_ptr)
        {
            using T_IN = std::remove_pointer_t<decltype(type_ptr)>;
            using T_OUT = T_IN;
            T_OUT* residualOut = reinterpret_cast<T_OUT*>(params.residualOut);
            T_OUT* output = reinterpret_cast<T_OUT*>(params.output);
            T_IN* bufferInput = reinterpret_cast<T_IN*>(params.bufferPtrLocal);
            T_IN const* gamma = reinterpret_cast<T_IN const*>(params.gamma);
            T_IN const* residualIn = reinterpret_cast<T_IN const*>(params.residualIn);
            if (rnUseCGA)
            {
                RUN_RMSNORM_KERNEL(T_IN, T_OUT, 1);
            }
            else
            {
                switch (rnLoadsPerThread)
                {
                case 1: RUN_RMSNORM_KERNEL(T_IN, T_OUT, 1); break;
                case 2: RUN_RMSNORM_KERNEL(T_IN, T_OUT, 2); break;
                case 3: RUN_RMSNORM_KERNEL(T_IN, T_OUT, 3); break;
                case 4: RUN_RMSNORM_KERNEL(T_IN, T_OUT, 4); break;
                case 5: RUN_RMSNORM_KERNEL(T_IN, T_OUT, 5); break;
                case 6: RUN_RMSNORM_KERNEL(T_IN, T_OUT, 6); break;
                case 7: RUN_RMSNORM_KERNEL(T_IN, T_OUT, 7); break;
                case 8: RUN_RMSNORM_KERNEL(T_IN, T_OUT, 8); break;
                default: return false;
                }
            }
            return true;
        };

        launched = (params.dType == nvinfer1::DataType::kFLOAT && dispatchRN((float*) nullptr))
            || (params.dType == nvinfer1::DataType::kBF16 && dispatchRN((__hip_bfloat16*) nullptr))
            || (params.dType == nvinfer1::DataType::kHALF && dispatchRN((__nv_half*) nullptr));
        if (!launched)
        {
            TLLM_CHECK_WITH_INFO(false, "[MNNVL AllReduceTwoShot] Failed to dispatch rmsnorm lamport kernel.");
        }
#undef RUN_RMSNORM_KERNEL
    }
}

} // namespace tensorrt_llm::kernels::mnnvl
