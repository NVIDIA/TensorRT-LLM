#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaBf16Fallbacks.cuh"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/customAllReduceUtils.h"
#include "tensorrt_llm/common/dataType.h"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/kernels/communicationKernels/customLowPrecisionAllReduceKernels.h"
#include <hip/hip_cooperative_groups.h>
#include <tuple>
#include <type_traits>

namespace tensorrt_llm::kernels
{

using tensorrt_llm::common::divUp;
using tensorrt_llm::common::roundUp;
using tensorrt_llm::common::cuda_max;
using tensorrt_llm::common::cuda_abs;

static StaticLowPrecisionBuffers static_tp2_buffers;
static StaticLowPrecisionBuffers static_tp4_buffers;
static StaticLowPrecisionBuffers static_tp8_buffers;

StaticLowPrecisionBuffers* getBufferForTpSize(size_t tpSize)
{
    if (tpSize == 2)
    {
        return &static_tp2_buffers;
    }
    else if (tpSize == 4)
    {
        return &static_tp4_buffers;
    }
    else if (tpSize == 8)
    {
        return &static_tp8_buffers;
    }
    else
    {
        TLLM_THROW("Unsupported tpSize for LowPrecisionCustomAllReduce");
    }
}

void initialize_static_lowprecision_buffers(int64_t* buffer, size_t tpSize)
{
    void* const* buffer_ptrs = reinterpret_cast<void* const*>(buffer);

    StaticLowPrecisionBuffers* static_buffers = getBufferForTpSize(tpSize);

    // Store pointers in static structure
    for (int i = 0; i < tpSize; ++i)
    {
        static_buffers->peer_comm_buffer_ptrs[i] = buffer_ptrs[i];
        static_buffers->peer_comm_buffer_ptrs[tpSize + i] = buffer_ptrs[tpSize + i];
        static_buffers->peer_barrier_ptrs_in[i] = reinterpret_cast<uint64_t*>(buffer_ptrs[2 * tpSize + i]);
        static_buffers->peer_barrier_ptrs_out[i] = reinterpret_cast<uint64_t*>(buffer_ptrs[3 * tpSize + i]);
    }

    constexpr int LOW_PRECISION_NUM_POINTERS_PER_RANK = 4;
    // Store the flag pointer
    int flag_offset = 1;
    static_buffers->flag_ptr = &buffer[LOW_PRECISION_NUM_POINTERS_PER_RANK * tpSize + flag_offset];

    static_buffers->initialized = true;
    static_buffers->tpSize = tpSize;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void lp_allreduce_st_flag_release(uint64_t const& flag, uint64_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("st.global.release.sys.b64 [%1], %0;" ::"l"(flag), "l"(flag_addr));
#else
    __threadfence_system();
    asm volatile("st.global.volatile.b64 [%1], %0;" ::"l"(flag), "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

static inline __device__ void lp_allreduce_ld_flag_acquire(uint64_t& flag, uint64_t* flag_addr)
{
#if __CUDA_ARCH__ >= 700
    asm volatile("ld.global.acquire.sys.b64 %0, [%1];" : "=l"(flag) : "l"(flag_addr));
#else
    asm volatile("ld.global.volatile.b64 %0, [%1];" : "=l"(flag) : "l"(flag_addr));
#endif
}

////////////////////////////////////////////////////////////////////////////////////////////////////

// Type Converter that packs data format to 128 bits data type
//
using PackedFloat = union
{
    int4 packed;
    float unpacked[4];
};

using PackedHalf = union
{
    int4 packed;
    // half2 unpacked[4];
    __half unpacked[8];
};

template <typename T>
struct PackedOn16Bytes
{
};

template <typename T, int Num>
struct PackedOnNum
{
};

template <>
struct PackedOn16Bytes<float>
{
    using Type = PackedFloat;
};

template <>
struct PackedOn16Bytes<half>
{
    using Type = PackedHalf;
};

using PackedINT8 = union
{
    int4 packed;
    int8_t unpacked[16];
};

using PackedINT8_8Bytes = union
{
    int2 packed;
    int8_t unpacked[8];
};

using PackedINT8_4Bytes = union
{
    int packed;
    int8_t unpacked[4];
};

template <>
struct PackedOn16Bytes<int8_t>
{
    using Type = PackedINT8;
};

template <>
struct PackedOnNum<int8_t, 8>
{
    using Type = PackedINT8_8Bytes;
};

template <>
struct PackedOnNum<int8_t, 4>
{
    using Type = PackedINT8_4Bytes;
};

#ifdef ENABLE_BF16
using PackedBFloat16 = union
{
    int4 packed;
    //__hip_bfloat162 unpacked[4];
    __hip_bfloat16 unpacked[8];
};

template <>
struct PackedOn16Bytes<__hip_bfloat16>
{
    using Type = PackedBFloat16;
};
#endif

#ifdef ENABLE_FP8
using PackedFloat8E4m3 = union
{
    int4 packed;
    __hip_fp8_e4m3_fnuz unpacked[16];
};

using PackedFloat8E4m3_8Bytes = union
{
    int2 packed;
    __hip_fp8_e4m3_fnuz unpacked[8];
};

using PackedFloat8E4m3_4Bytes = union
{
    int packed;
    __hip_fp8_e4m3_fnuz unpacked[4];
};

template <>
struct PackedOn16Bytes<__hip_fp8_e4m3_fnuz>
{
    using Type = PackedFloat8E4m3;
};

template <>
struct PackedOnNum<__hip_fp8_e4m3_fnuz, 8>
{
    using Type = PackedFloat8E4m3_8Bytes;
};

template <>
struct PackedOnNum<__hip_fp8_e4m3_fnuz, 4>
{
    using Type = PackedFloat8E4m3_4Bytes;
};
#endif

template <int num>
struct LowPrecisionIntPack
{
};

template <>
struct LowPrecisionIntPack<4>
{
    using Type = int;
};

template <>
struct LowPrecisionIntPack<8>
{
    using Type = int2;
};

template <>
struct LowPrecisionIntPack<16>
{
    using Type = int4;
};

__inline__ __device__ void multi_gpu_barrier(
    uint64_t** signals, const uint64_t flag, const size_t rank, const size_t world_size, int const tidx, int const bidx)
{
    // At the end of the function, we now that has least block 0 from all others GPUs have reached that point.
    uint64_t volatile* my_signals = signals[rank];
    if (tidx < world_size)
    {
        // The 1st block notifies the other ranks.
        if (bidx == 0)
        {
            signals[tidx][rank] = flag;
        }

        // Busy-wait until all ranks are ready.
        while (my_signals[tidx] != flag)
        {
        }
    }

    // Make sure we can move on...
    __syncthreads();
}

__device__ __forceinline__ void st_global_release(int4 const& val, int4* addr)
{
    asm volatile("st.release.global.sys.v4.b32 [%4], {%0, %1, %2, %3};" ::"r"(val.x), "r"(val.y), "r"(val.z),
        "r"(val.w), "l"(addr));
}

__device__ __forceinline__ int4 ld_global_acquire(int4* addr)
{
    int4 val;
    asm volatile("ld.acquire.global.sys.v4.b32 {%0, %1, %2, %3}, [%4];"
                 : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                 : "l"(addr));
    return val;
}

__device__ __forceinline__ void st_global_volatile(int4 const& val, int4* addr)
{
    asm volatile("st.volatile.global.v4.b32 [%4], {%0, %1, %2, %3};" ::"r"(val.x), "r"(val.y), "r"(val.z), "r"(val.w),
        "l"(addr));
}

__device__ __forceinline__ int4 ld_global_volatile(int4* addr)
{
    int4 val;
    asm volatile("ld.volatile.global.v4.b32 {%0, %1, %2, %3}, [%4];"
                 : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)
                 : "l"(addr));
    return val;
}

__device__ __forceinline__ void fence_acq_rel_sys()
{
    asm volatile("fence.acq_rel.sys;" ::: "memory");
}

template <typename T>
__device__ __forceinline__ uintptr_t cvta_to_global(T* ptr)
{
    return (uintptr_t) __cvta_generic_to_global(ptr);
}

__device__ __forceinline__ uint64_t ld_volatile_global(uint64_t* ptr)
{
    uint64_t ans;
    asm("ld.volatile.global.u64 %0, [%1];" : "=l"(ans) : "l"(cvta_to_global(ptr)));
    return ans;
}

__device__ __forceinline__ void wait_send_peer(uint64_t local_flag, uint64_t* peer_flag_ptr)
{
    uint64_t peer_flag = ld_volatile_global(peer_flag_ptr);
    while (local_flag - peer_flag >= LP_ALLREDUCE_BUFFER_CHUNKS)
    {
        peer_flag = ld_volatile_global(peer_flag_ptr);
    }
    return;
}

__device__ __forceinline__ void wait_recv_peer(uint64_t local_flag, uint64_t* peer_flag_ptr)
{
    uint64_t peer_flag = ld_volatile_global(peer_flag_ptr);

    while (local_flag >= peer_flag)
    {
        peer_flag = ld_volatile_global(peer_flag_ptr);
    }
    return;
}

__device__ __forceinline__ void notify_peer(uint64_t* peer_flag_ptr)
{
    asm volatile("st.relaxed.sys.global.u64 [%0], %1;" ::"l"(cvta_to_global(peer_flag_ptr)), "l"(uint64_t(1))
                 : "memory");
    return;
}

__device__ __forceinline__ void notify_peer_with_value_relax(uint64_t* peer_flag_ptr, uint64_t value)
{
    asm volatile("st.relaxed.sys.global.u64 [%0], %1;" ::"l"(cvta_to_global(peer_flag_ptr)), "l"(value) : "memory");
    return;
}

__device__ __forceinline__ void notify_peer_with_value(uint64_t* peer_flag_ptr, uint64_t value)
{
    *peer_flag_ptr = value;
    return;
}

__device__ float warp_reduce_max(float val)
{
    val = cuda_max(__shfl_xor_sync(~0, val, 16), val);
    val = cuda_max(__shfl_xor_sync(~0, val, 8), val);
    val = cuda_max(__shfl_xor_sync(~0, val, 4), val);
    val = cuda_max(__shfl_xor_sync(~0, val, 2), val);
    val = cuda_max(__shfl_xor_sync(~0, val, 1), val);
    return val;
}

template <typename QUANTIZE_T>
struct QuantMaxValue;

template <>
struct QuantMaxValue<int8_t>
{
    static constexpr float value = 127.0f;
};

template <>
struct QuantMaxValue<__hip_fp8_e4m3_fnuz>
{
    static constexpr float value = 448.0f;
};

template <int32_t RANKS_PER_NODE, typename T_IN, typename T_OUT>
__global__ void lowPrecisionPreprocessKernel(
    const T_IN* __restrict__ input, size_t elts_per_rank_in, size_t elts_per_rank_out, T_OUT* __restrict__ output)
{
    constexpr float QUANT_MAX = QuantMaxValue<T_OUT>::value;
    constexpr int32_t output_rounds = sizeof(T_IN) / sizeof(T_OUT);
    constexpr int32_t elts_per_thread = sizeof(int4) / sizeof(T_OUT);
    constexpr int32_t elts_per_round = sizeof(int4) / sizeof(T_IN);
    constexpr int32_t elts_per_warp_per_round = elts_per_round * WARP_SIZE;
    constexpr int32_t NUM_ELTS_PER_WARP_IN = (WARP_SIZE - 1) * elts_per_thread;
    constexpr int32_t NUM_ELTS_PER_WARP_OUT = WARP_SIZE * elts_per_thread;
    using PackedInputType = typename PackedOn16Bytes<T_IN>::Type;
    using PackedOutputType = typename PackedOnNum<T_OUT, elts_per_round>::Type;

    using PackedInputIntType = typename LowPrecisionIntPack<sizeof(int4)>::Type;
    using PackedOutputIntType = typename LowPrecisionIntPack<elts_per_round>::Type;

    const int32_t target_rank = blockIdx.x / (gridDim.x / RANKS_PER_NODE);
    const int32_t local_bid = blockIdx.x % (gridDim.x / RANKS_PER_NODE);

    input += elts_per_rank_in * target_rank;
    output += elts_per_rank_out * target_rank;

    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t wid = threadIdx.x / WARP_SIZE;

    PackedInputType vals[output_rounds];
    size_t start_in = NUM_ELTS_PER_WARP_IN * LP_ALLREDUCE_WARP_NUM_PER_BLOCK * local_bid + wid * NUM_ELTS_PER_WARP_IN;
    size_t start_out
        = NUM_ELTS_PER_WARP_OUT * LP_ALLREDUCE_WARP_NUM_PER_BLOCK * local_bid + wid * NUM_ELTS_PER_WARP_OUT;

#pragma unroll
    for (int32_t i = 0; i < output_rounds; ++i)
    {
        int32_t local_offset = lane_id * elts_per_round + elts_per_warp_per_round * i;
        int32_t global_offset = start_in + local_offset;
        if (local_offset < NUM_ELTS_PER_WARP_IN && global_offset < elts_per_rank_in)
        {
            vals[i].packed = *reinterpret_cast<PackedInputIntType const*>(input + start_in + local_offset);
        }
        else
        {
#pragma unroll
            for (int j = 0; j < elts_per_round; j++)
            {
                vals[i].unpacked[j] = 0.0f;
            }
        }
    }

    // Calculate scaling factor
    float scalar = 0;
    for (int32_t i = 0; i < output_rounds; ++i)
    {
#pragma unroll
        for (int32_t j = 0; j < elts_per_round; ++j)
        {
            scalar = cuda_max(cuda_abs((float) (vals[i].unpacked[j])), scalar);
        }
    }

    scalar = warp_reduce_max(scalar);
    if (scalar != 0.0f)
    {
        scalar = QUANT_MAX / scalar;
    }

    // Quantize and write output
    PackedOutputType output_vals[output_rounds];
    for (int32_t i = 0; i < output_rounds; ++i)
    {
        int32_t local_write_offset = lane_id * elts_per_round + elts_per_warp_per_round * i;
        if (local_write_offset < NUM_ELTS_PER_WARP_IN)
        {
#pragma unroll
            for (int32_t j = 0; j < elts_per_round; ++j)
            {
                float out_val = vals[i].unpacked[j];
                if (scalar != 0.0f)
                {
                    out_val *= scalar;
                }
                output_vals[i].unpacked[j] = static_cast<T_OUT>(out_val);
            }
        }
        else if (local_write_offset == NUM_ELTS_PER_WARP_IN)
        {
            *(reinterpret_cast<float*>(&output_vals[i])) = scalar;
        }
    }

#pragma unroll
    for (int32_t i = 0; i < output_rounds; ++i)
    {

        int32_t local_write_offset = lane_id * elts_per_round + elts_per_warp_per_round * i;
        *reinterpret_cast<PackedOutputIntType*>(output + start_out + local_write_offset) = output_vals[i].packed;
    }
}

template <int32_t RANKS_PER_NODE, typename T_IN>
__device__ void lowPrecisionTwoShotFirstStageKernel(int32_t myrank, size_t elts_per_rank, T_IN** input, float* smem)
{
    constexpr float QUANT_MAX = QuantMaxValue<T_IN>::value;
    constexpr int32_t elts_per_thread = sizeof(int4) / sizeof(T_IN);
    constexpr int32_t NUM_ELTS_PER_WARP_IN = WARP_SIZE * elts_per_thread;

    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t bid = blockIdx.x;
    const int32_t wid = threadIdx.x / WARP_SIZE;
    const size_t in_start
        = (bid * LP_ALLREDUCE_WARP_NUM_PER_BLOCK + wid) * NUM_ELTS_PER_WARP_IN + lane_id * elts_per_thread;

    // Packed data type for comms
    using PackedType = typename PackedOn16Bytes<T_IN>::Type;
    float* smem_scalar_ptr = &smem[RANKS_PER_NODE * wid];
    const size_t rank_offset = elts_per_rank * myrank;

    for (size_t local_offset = in_start; local_offset < elts_per_rank;
         local_offset += gridDim.x * blockDim.x * elts_per_thread)
    {
        float sums[elts_per_thread];
#pragma unroll
        for (int32_t ii = 0; ii < elts_per_thread; ++ii)
        {
            sums[ii] = 0;
        }

        // Read, dequantize and reduce sum
        {
            PackedType vals[RANKS_PER_NODE];
#pragma unroll
            for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&input[ii][local_offset + rank_offset]);
            }

            if (lane_id == (WARP_SIZE - 1))
            {
#pragma unroll
                for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
                {
                    float* tmp_scalar = (float*) (&(vals[ii]));
                    smem_scalar_ptr[ii] = tmp_scalar[0];
                }
            }
            __syncwarp();

            if (lane_id < (WARP_SIZE - 1))
            {
                // Sum the values from the different ranks
                for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
                {
#pragma unroll
                    for (int32_t jj = 0; jj < elts_per_thread; ++jj)
                    {
                        if (smem_scalar_ptr[ii] != 0)
                        {
                            sums[jj] += (float) (vals[ii].unpacked[jj]) / smem_scalar_ptr[ii];
                        }
                        else
                        {
                            sums[jj] += (float) (vals[ii].unpacked[jj]);
                        }
                    }
                }
            }
        }

        // Quantize and write back results
        {
            float scalar = 0;

            if (lane_id < (WARP_SIZE - 1))
            {
#pragma unroll
                for (int32_t ii = 0; ii < elts_per_thread; ++ii)
                {
                    scalar = cuda_max(cuda_abs(sums[ii]), scalar);
                }
            }

            scalar = warp_reduce_max(scalar);

            if (scalar != 0.0f)
            {
                scalar = (QUANT_MAX) / scalar;
            }

            PackedType tmp_val;
            if (lane_id < (WARP_SIZE - 1))
            {
#pragma unroll
                for (int32_t ii = 0; ii < elts_per_thread; ++ii)
                {
                    float tmp = sums[ii];
                    if (scalar != 0.0f)
                    {
                        tmp *= scalar;
                    }
                    tmp_val.unpacked[ii] = static_cast<T_IN>(tmp);
                }
            }
            else
            {
                ((float*) (&tmp_val))[0] = scalar;
            }

            *reinterpret_cast<int4*>(input[0] + local_offset + rank_offset) = tmp_val.packed;
        }
    }
}

template <int32_t RANKS_PER_NODE, typename T_IN, typename T_OUT>
__device__ void lowPrecisionTwoShotSecondStageKernel(size_t input_elts_per_rank, size_t output_elts_per_rank,
    T_IN** input, T_OUT* output, float* smem, int32_t* dst_rank)
{
    constexpr int32_t elts_per_thread = sizeof(int4) / sizeof(T_IN);
    constexpr int32_t output_rounds = sizeof(T_OUT) / sizeof(T_IN);
    constexpr int32_t depack_num = elts_per_thread / output_rounds;

    constexpr int32_t NUM_ELTS_PER_WARP_IN = WARP_SIZE * elts_per_thread;
    constexpr int32_t NUM_ELTS_PER_WARP_OUT = (WARP_SIZE - 1) * elts_per_thread;

    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t bid = blockIdx.x;
    const int32_t wid = threadIdx.x / WARP_SIZE;

    const size_t in_start
        = (bid * LP_ALLREDUCE_WARP_NUM_PER_BLOCK + wid) * NUM_ELTS_PER_WARP_IN + lane_id * elts_per_thread;
    const size_t out_start
        = (bid * LP_ALLREDUCE_WARP_NUM_PER_BLOCK + wid) * NUM_ELTS_PER_WARP_OUT + lane_id * elts_per_thread;

    float* smem_scalar_ptr = &smem[RANKS_PER_NODE * wid];

    using PackedInType = typename PackedOn16Bytes<T_IN>::Type;
    using PackedOutType = typename PackedOn16Bytes<T_OUT>::Type;

    PackedInType vals[RANKS_PER_NODE];

    for (size_t input_offset = in_start, output_offset = out_start; input_offset < input_elts_per_rank;
         input_offset += gridDim.x * LP_ALLREDUCE_WARP_NUM_PER_BLOCK * NUM_ELTS_PER_WARP_IN,
                output_offset += gridDim.x * LP_ALLREDUCE_WARP_NUM_PER_BLOCK * NUM_ELTS_PER_WARP_OUT)
    {
#pragma unroll
        for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            size_t tmp_offset = dst_rank[ii] * input_elts_per_rank + input_offset;
            if (input_offset < input_elts_per_rank)
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&input[ii][tmp_offset]);
            }
        }

        if (lane_id == (WARP_SIZE - 1))
        {
#pragma unroll
            for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
            {
                float* tmp_scalar = (float*) (&(vals[ii]));
                smem_scalar_ptr[ii] = tmp_scalar[0];
            }
        }
        __syncwarp();

        for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            float scale = smem_scalar_ptr[ii];
            size_t tmp_output_offset = dst_rank[ii] * output_elts_per_rank + output_offset;

            if (output_offset < output_elts_per_rank)
            {
                if (lane_id < (WARP_SIZE - 1))
                {
                    for (int32_t jj = 0; jj < output_rounds; ++jj)
                    {
                        PackedOutType tmp_output;

#pragma unroll
                        for (int32_t kk = 0; kk < depack_num; kk++)
                        {
                            float tmp = (float) (vals[ii].unpacked[kk + jj * depack_num]);
                            if (scale != 0.0f)
                            {
                                tmp /= scale;
                            }
                            tmp_output.unpacked[kk] = static_cast<T_OUT>(tmp);
                        }

                        *reinterpret_cast<PackedOutType*>(output + tmp_output_offset + jj * depack_num) = tmp_output;
                    }
                }
            }
        }
    }
}

template <typename T, typename QUANT_T, int32_t RANKS_PER_NODE>
static __global__ void lowPrecisionTwoShotAllReduceKernel(LowPrecisionAllReduceParams params)
{
    const int32_t bidx = blockIdx.x;
    const int32_t tidx = threadIdx.x;

    extern __shared__ float smem[];

    multi_gpu_barrier(params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);

    // The source pointers. Distributed round-robin for the different warps.
    QUANT_T* src_d[RANKS_PER_NODE];
    // The destination ranks for round-robin gathering
    int32_t dst_rank[RANKS_PER_NODE];

#pragma unroll
    for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
    {
        int32_t rank = (params.local_rank + ii) % RANKS_PER_NODE;
        src_d[ii] = reinterpret_cast<QUANT_T*>(params.peer_comm_buffer_ptrs[rank]);
        dst_rank[ii] = rank;
    }

    lowPrecisionTwoShotFirstStageKernel<RANKS_PER_NODE, QUANT_T>(
        params.local_rank, params.buffer_elts_per_rank, src_d, smem);

    // Sync threads to make sure all block threads have the sums
    __syncthreads();

    // Barriers among the blocks with the same idx (release-acquire semantics)
    if (tidx < RANKS_PER_NODE)
    {
        // The all blocks notifies the other ranks.
        uint32_t flag_block_offset = RANKS_PER_NODE + bidx * RANKS_PER_NODE;
        lp_allreduce_st_flag_release(
            params.barrier_flag, params.peer_barrier_ptrs_in[tidx] + flag_block_offset + params.local_rank);

        // Busy-wait until all ranks are ready.
        uint64_t rank_barrier = 0;
        uint64_t* peer_barrier_d = params.peer_barrier_ptrs_in[params.local_rank] + flag_block_offset + tidx;
        do
        {
            lp_allreduce_ld_flag_acquire(rank_barrier, peer_barrier_d);
        } while (rank_barrier != params.barrier_flag);
    }

    __syncthreads();

    // Do allgather and dequantize
    float* smem_allgather = smem + (RANKS_PER_NODE * LP_ALLREDUCE_WARP_NUM_PER_BLOCK);
    lowPrecisionTwoShotSecondStageKernel<RANKS_PER_NODE, QUANT_T, T>(params.buffer_elts_per_rank, params.elts_per_rank,
        src_d, reinterpret_cast<T*>(params.local_output_buffer_ptr), smem_allgather, dst_rank);
}

template <typename T_IN, typename T_OUT>
__global__ void lowPrecisionHierPreprocessKernel(
    const T_IN* __restrict__ input, size_t n_in, T_OUT* __restrict__ output)
{
    constexpr float QUANT_MAX = QuantMaxValue<T_OUT>::value;
    constexpr int32_t output_rounds = sizeof(T_IN) / sizeof(T_OUT);
    constexpr int32_t elts_per_thread = sizeof(int4) / sizeof(T_OUT);
    constexpr int32_t elts_per_round = sizeof(int4) / sizeof(T_IN);
    constexpr int32_t elts_per_warp_per_round = elts_per_round * WARP_SIZE;
    constexpr int32_t NUM_ELTS_PER_WARP_IN = (WARP_SIZE - 1) * elts_per_thread;
    constexpr int32_t NUM_ELTS_PER_WARP_OUT = WARP_SIZE * elts_per_thread;

    using PackedInputType = typename PackedOn16Bytes<T_IN>::Type;
    using PackedOutputType = typename PackedOnNum<T_OUT, elts_per_round>::Type;
    using PackedInputIntType = typename LowPrecisionIntPack<16>::Type;
    using PackedOutputIntType = typename LowPrecisionIntPack<elts_per_round>::Type;

    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t wid = threadIdx.x / WARP_SIZE;
    PackedInputType vals[output_rounds];

    for (size_t start = blockIdx.x * LP_ALLREDUCE_WARP_NUM_PER_BLOCK + wid; start * NUM_ELTS_PER_WARP_IN < n_in;
         start += LP_ALLREDUCE_WARP_NUM_PER_BLOCK * gridDim.x)
    {
        int32_t read_rounds = 0;
        int32_t local_n_in = (n_in - start * NUM_ELTS_PER_WARP_IN) > NUM_ELTS_PER_WARP_IN
            ? NUM_ELTS_PER_WARP_IN
            : (n_in - start * NUM_ELTS_PER_WARP_IN);
        if (local_n_in <= 0)
        {
            return;
        }

#pragma unroll
        for (int32_t i = 0; i < output_rounds; ++i)
        {
            int32_t local_offset = lane_id * elts_per_round + elts_per_warp_per_round * i;
            if (local_offset < local_n_in)
            {
                vals[i].packed
                    = *reinterpret_cast<PackedInputIntType const*>(input + start * NUM_ELTS_PER_WARP_IN + local_offset);
                read_rounds++;
            }
            else
            {
#pragma unroll
                for (int j = 0; j < elts_per_round; j++)
                {
                    vals[i].unpacked[j] = 0.0f;
                }
            }
        }

        // Calculate scaling factor
        float scalar = 0;
        for (int32_t i = 0; i < read_rounds; ++i)
        {
#pragma unroll
            for (int32_t j = 0; j < elts_per_round; ++j)
            {
                scalar = cuda_max(cuda_abs((float) (vals[i].unpacked[j])), scalar);
            }
        }

        scalar = warp_reduce_max(scalar);
        if (scalar != 0.0f)
        {
            scalar = QUANT_MAX / scalar;
        }

        // Quantize and write output
        PackedOutputType output_vals[output_rounds];
        for (int32_t i = 0; i < output_rounds; ++i)
        {
            int32_t local_write_offset = lane_id * elts_per_round + elts_per_warp_per_round * i;
            if (local_write_offset < NUM_ELTS_PER_WARP_IN)
            {
#pragma unroll
                for (int32_t j = 0; j < elts_per_round; ++j)
                {
                    float out_val = vals[i].unpacked[j];
                    if (scalar != 0.0f)
                    {
                        out_val *= scalar;
                    }
                    output_vals[i].unpacked[j] = static_cast<T_OUT>(out_val);
                }
            }
            else if (local_write_offset == NUM_ELTS_PER_WARP_IN)
            {
                *(reinterpret_cast<float*>(&output_vals[i])) = scalar;
            }
        }

#pragma unroll
        for (int32_t i = 0; i < output_rounds; ++i)
        {
            int32_t local_write_offset = lane_id * elts_per_round + elts_per_warp_per_round * i;
            *reinterpret_cast<PackedOutputIntType*>(output + start * NUM_ELTS_PER_WARP_OUT + local_write_offset)
                = output_vals[i].packed;
        }
    }
}

template <int32_t RANKS_PER_NODE, typename T>
__device__ void hierReduceWithQdq(
    LowPrecisionAllReduceParams params, T** input, T* output, int64_t start_offset, int64_t length, float* smem)
{
    // Constants
    constexpr float QUANT_MAX = QuantMaxValue<T>::value;
    constexpr int32_t elts_per_thread = sizeof(int4) / sizeof(T);

    // Thread indices
    const int32_t lane_id = threadIdx.x % WARP_SIZE;
    const int32_t wid = threadIdx.x / WARP_SIZE;
    const size_t start = threadIdx.x * elts_per_thread;

    // Packed data type for comms
    using PackedType = typename PackedOn16Bytes<T>::Type;
    float* smem_scalar_ptr = &smem[RANKS_PER_NODE * wid];

    for (size_t index = start; index < length; index += LP_ALLREDUCE_DEFAULT_BLOCK_SIZE * elts_per_thread)
    {
        // Initialize sum array
        float sums[elts_per_thread];
#pragma unroll
        for (int32_t ii = 0; ii < elts_per_thread; ++ii)
        {
            sums[ii] = 0;
        }

        // Load values from different ranks and dequantize
        {
            PackedType vals[RANKS_PER_NODE];

#pragma unroll
            for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
            {
                vals[ii].packed = *reinterpret_cast<int4 const*>(&input[ii][start_offset + index]);
            }

            if (lane_id == (WARP_SIZE - 1))
            {
#pragma unroll
                for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
                {
                    float* tmp_scalar = (float*) (&(vals[ii]));
                    smem_scalar_ptr[ii] = tmp_scalar[0];
                }
            }
            __syncwarp();

            if (lane_id < (WARP_SIZE - 1))
            {
                for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
                {
#pragma unroll
                    for (int32_t jj = 0; jj < elts_per_thread; ++jj)
                    {
                        if (smem_scalar_ptr[ii] != 0)
                        {
                            sums[jj] += (float) (vals[ii].unpacked[jj]) / smem_scalar_ptr[ii];
                        }
                        else
                        {
                            sums[jj] += (float) (vals[ii].unpacked[jj]);
                        }
                    }
                }
            }
        }

        // Quantize results and write output
        {
            float scalar = 0;

            if (lane_id < (WARP_SIZE - 1))
            {
#pragma unroll
                for (int32_t ii = 0; ii < elts_per_thread; ++ii)
                {
                    scalar = cuda_max(cuda_abs(sums[ii]), scalar);
                }
            }

            scalar = warp_reduce_max(scalar);

            if (scalar != 0.0f)
            {
                scalar = QUANT_MAX / scalar;
            }

            PackedType tmp_val;

            if (lane_id < (WARP_SIZE - 1))
            {
#pragma unroll
                for (int32_t ii = 0; ii < elts_per_thread; ++ii)
                {
                    float tmp = sums[ii];
                    if (scalar != 0.0f)
                    {
                        tmp *= scalar;
                    }
                    tmp_val.unpacked[ii] = (T) tmp;
                }
            }
            else
            {
                ((float*) (&tmp_val))[0] = scalar;
            }

            *reinterpret_cast<int4*>(&output[threadIdx.x * elts_per_thread]) = tmp_val.packed;
        }
    }
}

template <int32_t RANKS_PER_NODE, typename T_IN, typename T_OUT>
__device__ void hierAllgatherWithDq(LowPrecisionAllReduceParams params, T_IN** input, T_OUT* output,
    size_t input_offset, int32_t global_iter, int32_t length, int32_t blocks_per_stage, float* smem)
{
    // Constants and thread indices
    constexpr int32_t elts_per_thread = sizeof(int4) / sizeof(T_IN);
    constexpr int32_t output_rounds = sizeof(T_OUT) / sizeof(T_IN);
    constexpr int32_t depack_num = elts_per_thread / output_rounds;

    const int32_t bidx = blockIdx.x;
    const int32_t tidx = threadIdx.x;
    const int32_t lane_id = tidx % WARP_SIZE;
    const int32_t wid = tidx / WARP_SIZE;
    const int32_t start = tidx * elts_per_thread;

    const int32_t OUTPUT_ELEMENT_PER_WARP = (WARP_SIZE - 1) * elts_per_thread;
    const int32_t OUTPUT_ELEMENT_PER_BLOCK = OUTPUT_ELEMENT_PER_WARP * LP_ALLREDUCE_WARP_NUM_PER_BLOCK;

    using PackedType = typename PackedOn16Bytes<T_IN>::Type;
    using PackedOutputType = typename PackedOn16Bytes<T_OUT>::Type;
    const int32_t numa_rank = params.numa_rank;

    PackedType vals[RANKS_PER_NODE];
    float* smem_scalar_ptr = &smem[RANKS_PER_NODE * wid];

    for (size_t index = start; index < length; index += LP_ALLREDUCE_DEFAULT_BLOCK_SIZE * elts_per_thread)
    {
#pragma unroll
        for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            vals[ii].packed = *reinterpret_cast<int4 const*>(&input[ii][input_offset + index]);
        }

#pragma unroll
        for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            if (lane_id == WARP_SIZE - 1)
            {
                float* tmp_scalar = (float*) (&(vals[ii]));
                smem_scalar_ptr[ii] = tmp_scalar[0];
            }
        }
        __syncwarp();

        const size_t elts_total = params.elts_total;

        for (int32_t ii = 0; ii < RANKS_PER_NODE; ++ii)
        {
            float scale = smem_scalar_ptr[ii];
            size_t offset_global = global_iter * blocks_per_stage * RANKS_PER_NODE * OUTPUT_ELEMENT_PER_BLOCK;

            int32_t tmp_rank = (numa_rank + ii) % RANKS_PER_NODE;
            size_t offset_local = offset_global + (bidx % blocks_per_stage) * RANKS_PER_NODE * OUTPUT_ELEMENT_PER_BLOCK
                + tmp_rank * OUTPUT_ELEMENT_PER_BLOCK + wid * OUTPUT_ELEMENT_PER_WARP + lane_id * elts_per_thread;
            bool need_write = elts_total > offset_local;

            if (lane_id < WARP_SIZE - 1 && need_write)
            {
                for (int32_t jj = 0; jj < output_rounds; ++jj)
                {
                    PackedOutputType tmp_output;

#pragma unroll
                    for (int32_t kk = 0; kk < depack_num; kk++)
                    {
                        float tmp = (float) (vals[ii].unpacked[kk + jj * depack_num]);
                        if (scale != 0)
                        {
                            tmp /= scale;
                        }
                        ((T_OUT*) (&tmp_output))[kk] = (T_OUT) tmp;
                    }

                    *reinterpret_cast<int4*>(&reinterpret_cast<T_OUT*>(output)[offset_local + jj * depack_num])
                        = *reinterpret_cast<int4*>(&tmp_output);
                }
            }
        }
    }
}

template <typename T, typename QUANT_T, int RANKS_PER_NODE>
static __global__ __launch_bounds__(512, 1) void lowPrecisionTwoShotHierAllReduceKernel(
    LowPrecisionAllReduceParams params)
{

    // The block index.
    int const bidx = blockIdx.x;
    // The thread index with the block.
    int const tidx = threadIdx.x;
    // The block num
    int const block_num = gridDim.x;
    int const duplicate = LP_ALLREDUCE_BUFFER_DUPLICATE;
    // this algorithm have 3 stages , so for one stage, have 1/3's block num
    int const block_num_per_stage = block_num / LP_ALLREDUCE_HIER_STAGE_NUM;

    // The number of elements packed into one for comms
    constexpr int elts_per_thread = sizeof(int4) / sizeof(QUANT_T);
    constexpr int ELTS_PER_BLOCK = elts_per_thread * LP_ALLREDUCE_DEFAULT_BLOCK_SIZE;

    extern __shared__ float smem[];

    multi_gpu_barrier(params.peer_barrier_ptrs_in, params.barrier_flag, params.local_rank, RANKS_PER_NODE, tidx, bidx);
    // Packed data type for comms
    using PackedType = typename PackedOn16Bytes<QUANT_T>::Type;

    if (bidx < block_num_per_stage)
    {
        // reduce-scatter inside NUMA
        int local_bid = bidx % block_num_per_stage;
        uint64_t send_flag = *params.rs_send_flags[local_bid];
        QUANT_T* src_d[LP_ALLREDUCE_RANKS_PER_NUMA];
        QUANT_T* dst = reinterpret_cast<QUANT_T*>(params.rs_buffers[local_bid]);

        // The destination ranks for round-robin gathering
#pragma unroll
        for (int ii = 0; ii < LP_ALLREDUCE_RANKS_PER_NUMA; ++ii)
        {
            int numa_rank = (params.numa_rank + ii) % LP_ALLREDUCE_RANKS_PER_NUMA;
            src_d[ii] = reinterpret_cast<QUANT_T*>(params.inputs_inside_numa[numa_rank]);
        }

        int32_t index = 0;
        while (index < params.num_rounds_fence)
        {
            if (tidx < LP_ALLREDUCE_NUMA_NUM)
            {
                wait_send_peer(send_flag, params.rs_ack_flags[local_bid] + tidx);
            }
            __syncthreads();
            int const processed = index * duplicate;
            int const remaining = params.num_rounds - processed;
            int const transfer_times = min(duplicate, remaining);

            for (int i = 0; i < transfer_times; ++i)
            {
                int const global_iter = index * duplicate + i;

                int const chunk_idx = send_flag % LP_ALLREDUCE_BUFFER_CHUNKS;
                int const dst_offset = chunk_idx * ELTS_PER_BLOCK * duplicate + ELTS_PER_BLOCK * i;
                int const global_per_tier = block_num_per_stage * LP_ALLREDUCE_RANKS_PER_NUMA * ELTS_PER_BLOCK;
                int const rank_offset = LP_ALLREDUCE_RANKS_PER_NUMA * ELTS_PER_BLOCK;
                const size_t global_offset
                    = global_iter * global_per_tier + local_bid * rank_offset + params.numa_rank * ELTS_PER_BLOCK;
                hierReduceWithQdq<LP_ALLREDUCE_RANKS_PER_NUMA, QUANT_T>(
                    params, src_d, dst + dst_offset, global_offset, ELTS_PER_BLOCK, smem);
            }

            __syncthreads();
            send_flag++;
            if (tidx == 0)
            {
                __threadfence_system();
                notify_peer_with_value(params.rs_notify_remote_flags[local_bid], send_flag);
                notify_peer_with_value(params.rs_notify_local_flags[local_bid], send_flag);
            }
            index++;
        }
        if (tidx == 0)
        {
            *params.rs_send_flags[local_bid] = send_flag;
        }
        return;
    }

    else if (bidx >= block_num_per_stage && bidx < block_num_per_stage * 2)
    {
        // partial allreduce cross NUMA
        int local_bid = bidx % block_num_per_stage;
        uint64_t send_flag = *params.ar_send_flags[local_bid];
        // 2 is all
        QUANT_T* src_d[LP_ALLREDUCE_NUMA_NUM];
        QUANT_T* dst = reinterpret_cast<QUANT_T*>(params.ar_buffers[local_bid]);
        src_d[0] = reinterpret_cast<QUANT_T*>(params.rs_buffers[local_bid]);
        src_d[1] = reinterpret_cast<QUANT_T*>(params.ar_peer_buffers_cross_numa[local_bid]);

        int32_t index = 0;
        while (index < params.num_rounds_fence)
        {
            if (tidx == 0)
            {
                wait_recv_peer(send_flag, params.rs_notify_local_flags[local_bid]);
                wait_recv_peer(send_flag, params.ar_ack_peer_rs_flags[local_bid]);
                wait_send_peer(send_flag, params.ar_ack_flags[local_bid]);
            }
            __syncthreads();

            int const processed = index * duplicate;
            int const remaining = params.num_rounds - processed;
            int const transfer_times = min(duplicate, remaining);

            int const chunk_idx = send_flag % LP_ALLREDUCE_BUFFER_CHUNKS;
            int const base_offset = chunk_idx * ELTS_PER_BLOCK * duplicate;

            for (int i = 0; i < transfer_times; ++i)
            {
                int const offset = base_offset + i * ELTS_PER_BLOCK;
                hierReduceWithQdq<LP_ALLREDUCE_NUMA_NUM, QUANT_T>(
                    params, src_d, dst + offset, offset, ELTS_PER_BLOCK, smem);
            }
            __syncthreads();

            send_flag++;
            if (tidx == 0)
            {
                __threadfence_system();
                notify_peer_with_value(params.ar_notify_rs_remote_flags[local_bid], send_flag);
                notify_peer_with_value(params.ar_notify_rs_local_flags[local_bid], send_flag);
                notify_peer_with_value(params.ar_notify_ag_flags[local_bid], send_flag);
            }
            index++;
        }
        if (tidx == 0)
        {
            *params.ar_send_flags[local_bid] = send_flag;
        }
        return;
    }
    else if (bidx >= block_num_per_stage * 2 && bidx < block_num_per_stage * 3)
    {
        // allgather inside NUMA
        int local_bid = bidx % block_num_per_stage;
        uint64_t send_flag = *params.ag_send_flags[local_bid];
        QUANT_T* src_d[LP_ALLREDUCE_RANKS_PER_NUMA];
        T* dst = reinterpret_cast<T*>(params.local_output_buffer_ptr);
#pragma unroll
        for (int ii = 0; ii < LP_ALLREDUCE_RANKS_PER_NUMA; ++ii)
        {
            int numa_rank = (params.numa_rank + ii) % LP_ALLREDUCE_RANKS_PER_NUMA;

            src_d[ii] = reinterpret_cast<QUANT_T*>(params.ag_peer_buffers_inside_numa[local_bid * 4 + numa_rank]);
        }

        int32_t index = 0;
        while (index < params.num_rounds_fence)
        {
            if (tidx == 0)
            {
                wait_recv_peer(send_flag, params.ar_notify_ag_flags[local_bid]);
            }

            __syncthreads();
            if (tidx < LP_ALLREDUCE_RANKS_PER_NUMA)
            {

                notify_peer_with_value_relax(
                    params.ag_notify_peer_inside_numa_flags[local_bid * LP_ALLREDUCE_RANKS_PER_NUMA + tidx],
                    send_flag + 1);
                wait_recv_peer(send_flag, params.ag_ack_peer_inside_numa_flags[local_bid] + tidx);
            }
            __syncthreads();

            int const processed = index * duplicate;
            int const remaining = params.num_rounds - processed;
            int const transfer_times = min(duplicate, remaining);

            int const chunk_idx = send_flag % LP_ALLREDUCE_BUFFER_CHUNKS;
            int const base_offset = chunk_idx * ELTS_PER_BLOCK * duplicate;

            for (int i = 0; i < transfer_times; ++i)
            {

                int const global_iter = processed + i;
                const size_t curr_offset = base_offset + i * ELTS_PER_BLOCK;

                hierAllgatherWithDq<LP_ALLREDUCE_RANKS_PER_NUMA, QUANT_T, T>(
                    params, src_d, dst, curr_offset, global_iter, ELTS_PER_BLOCK, block_num_per_stage, smem);
            }

            __syncthreads();

            send_flag++;
            if (tidx == 0)
            {
                notify_peer_with_value_relax(params.ar_ack_flags[local_bid], send_flag);
            }
            index++;
        }
        if (tidx == 0)
        {
            *params.ag_send_flags[local_bid] = send_flag;
        }
    }
    else
    {
        return;
    }
}

template <typename T, typename QUANT_T, int RANKS_PER_NODE>
void lowPrecisionAllReduceDispatchRanksPerNode(kernels::LowPrecisionAllReduceParams& params, hipStream_t stream)
{
    constexpr int qtype_elts_per_load = LP_ALLREDUCE_BYTES_PER_LOAD / sizeof(QUANT_T);
    constexpr int elts_per_block = qtype_elts_per_load * (LP_ALLREDUCE_WARPSIZE - 1) * LP_ALLREDUCE_WARP_NUM_PER_BLOCK;
    constexpr int elts_per_block_with_scale = qtype_elts_per_load * LP_ALLREDUCE_DEFAULT_BLOCK_SIZE;
    if (RANKS_PER_NODE <= 4)
    {

        int blocks_per_grid = LP_ALLREDUCE_MAX_BLOCKS * 2, threads_per_block = LP_ALLREDUCE_DEFAULT_BLOCK_SIZE;

        params.elts_per_rank = params.elts_total / RANKS_PER_NODE;
        params.rank_offset = params.rank * params.elts_per_rank;
        params.elts_per_block = elts_per_block;

        size_t num_rounds_per_rank = (params.elts_per_rank - 1) / elts_per_block + 1;
        size_t my_rank = params.local_rank;

        params.buffer_offset = my_rank * elts_per_block_with_scale * num_rounds_per_rank;
        params.buffer_elts_per_rank = elts_per_block_with_scale * num_rounds_per_rank;
        lowPrecisionPreprocessKernel<RANKS_PER_NODE, T, QUANT_T>
            <<<num_rounds_per_rank * RANKS_PER_NODE, threads_per_block, 0, stream>>>(
                (T const*) params.local_input_buffer_ptr, params.elts_per_rank, params.buffer_elts_per_rank,
                (QUANT_T*) params.peer_comm_buffer_ptrs[my_rank]);
        lowPrecisionTwoShotAllReduceKernel<T, QUANT_T, RANKS_PER_NODE><<<blocks_per_grid, threads_per_block,
            (LP_ALLREDUCE_WARP_NUM_PER_BLOCK * RANKS_PER_NODE) * sizeof(float) * 2, stream>>>(params);
    }
    else
    {
        int blocks_per_grid = LP_ALLREDUCE_MAX_BLOCKS, threads_per_block = LP_ALLREDUCE_DEFAULT_BLOCK_SIZE;
        params.num_rounds = (((params.elts_total - 1) / elts_per_block + 1) - 1) / LP_ALLREDUCE_MAX_RANKS_PER_NUMA
                / LP_ALLREDUCE_MAX_BLOCKS
            + 1;
        params.num_rounds_fence = (params.num_rounds - 1) / LP_ALLREDUCE_BUFFER_DUPLICATE + 1;
        blocks_per_grid = params.num_rounds < LP_ALLREDUCE_MAX_BLOCKS ? params.num_rounds : blocks_per_grid;

        size_t preprocess_blocks_per_grid = params.num_rounds * LP_ALLREDUCE_MAX_RANKS_PER_NUMA * blocks_per_grid;
        size_t my_rank = params.local_rank;
        blocks_per_grid *= LP_ALLREDUCE_HIER_STAGE_NUM; // 3 stages need more block

        lowPrecisionHierPreprocessKernel<T, QUANT_T><<<preprocess_blocks_per_grid, LP_ALLREDUCE_DEFAULT_BLOCK_SIZE,
            (LP_ALLREDUCE_WARP_NUM_PER_BLOCK) * sizeof(float), stream>>>((T const*) params.local_input_buffer_ptr,
            params.elts_total, (QUANT_T*) params.peer_comm_buffer_ptrs[my_rank]);
        lowPrecisionTwoShotHierAllReduceKernel<T, QUANT_T, RANKS_PER_NODE><<<blocks_per_grid, threads_per_block,
            (LP_ALLREDUCE_WARP_NUM_PER_BLOCK * RANKS_PER_NODE) * sizeof(float), stream>>>(params);
    }
}

template <typename T>
void lowPrecisionAllReduceDispatchType(kernels::LowPrecisionAllReduceParams& param, hipStream_t stream)
{
#ifdef ENABLE_FP8
    switch (param.ranks_per_node)
    {
    case 2: lowPrecisionAllReduceDispatchRanksPerNode<T, __hip_fp8_e4m3_fnuz, 2>(param, stream); break;
    case 4: lowPrecisionAllReduceDispatchRanksPerNode<T, __hip_fp8_e4m3_fnuz, 4>(param, stream); break;
    case 8: lowPrecisionAllReduceDispatchRanksPerNode<T, __hip_fp8_e4m3_fnuz, 8>(param, stream); break;
    default: TLLM_THROW("Custom LowPrecision all reduce only supported on {2, 4, 8} GPUs per node.");
    }
#else
    TLLM_THROW("Can't Use Low Precision Allreduce When Compile Without ENABLE_FP8");
#endif
}

std::vector<size_t> splitNumber(size_t number)
{
    std::vector<size_t> parts;
    size_t parts_num = number / LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE;
    size_t remain = number % LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE;
    if (parts_num == 0)
    {
        parts.push_back(remain);
    }
    else
    {
        if (remain == 0)
        {
            for (size_t i = 0; i < parts_num; ++i)
            {
                parts.push_back(LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE);
            }
        }
        else
        {
            for (size_t i = 0; i < parts_num - 1; ++i)
            {
                parts.push_back(LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE);
            }
            // if last remain part is small, will split a normal part, and fuse remain part to half normal
            // part
            if (remain < LP_ALLREDUCE_MIN_ELTS_THRESHOLD)
            {
                parts.push_back(LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE / 2 + remain);
                parts.push_back(LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE / 2);
            }
            else
            {
                parts.push_back(LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE);
                parts.push_back(remain);
            }
        }
    }
    return parts;
}

LowPrecisionAllReduceParams LowPrecisionAllReduceParams::deserialize(
    size_t tpSize, size_t tpRank, nvinfer1::DataType dataType, int token_num, int hidden_size)
{

    // Get appropriate static buffer
    StaticLowPrecisionBuffers* static_buffers = getBufferForTpSize(tpSize);

    // Check initialization
    if (!static_buffers->initialized || static_buffers->tpSize != tpSize)
    {
        TLLM_THROW("Static buffers for TP size %zu not initialized", tpSize);
    }

    // Use the stored flag pointer
    *(static_buffers->flag_ptr) += 1;

    TLLM_LOG_TRACE("AllReduceParams's flag value is %d", *(static_buffers->flag_ptr));
    uint64_t flag_value = *(static_buffers->flag_ptr);
    LowPrecisionAllReduceParams params;
    // Even plugins use ping buffers, odd plugins use pong.
    // That way, we don't need to wait for other GPUs to be done
    // before copying input tensor to workspace.
    auto const buffer_offset = (flag_value % 2 == 0) ? 0 : tpSize;

    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_comm_buffer_ptrs[i] = static_buffers->peer_comm_buffer_ptrs[buffer_offset + i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_in[i] = static_buffers->peer_barrier_ptrs_in[i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_out[i] = static_buffers->peer_barrier_ptrs_out[i];
    }
    // Assume that a single allreduce will not be divided into more than 64 allreduces of 64MB each,it is not very safe
    params.barrier_flag = flag_value;
    params.ranks_per_node = tpSize;
    params.local_rank = tpRank;

    return params;
}

LowPrecisionAllReduceParams LowPrecisionAllReduceParams::deserialize_hier(
    size_t tpSize, size_t tpRank, nvinfer1::DataType dataType, int token_num, int hidden_size)
{

    // Get appropriate static buffer
    StaticLowPrecisionBuffers* static_buffers = getBufferForTpSize(tpSize);

    // Check initialization
    if (!static_buffers->initialized || static_buffers->tpSize != tpSize)
    {
        TLLM_THROW("Static buffers for TP size %zu not initialized", tpSize);
    }

    // Use the stored flag pointer
    *(static_buffers->flag_ptr) += 1;

    TLLM_LOG_TRACE("AllReduceParams's flag value is %d", *(static_buffers->flag_ptr));
    uint64_t flag_value = *(static_buffers->flag_ptr);
    LowPrecisionAllReduceParams params;
    // Even plugins use ping buffers, odd plugins use pong.
    // That way, we don't need to wait for other GPUs to be done
    // before copying input tensor to workspace.
    auto const buffer_offset = (flag_value % 2 == 0) ? 0 : tpSize;

    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_comm_buffer_ptrs[i] = static_buffers->peer_comm_buffer_ptrs[buffer_offset + i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_in[i] = static_buffers->peer_barrier_ptrs_in[i];
    }
    for (int i = 0; i < tpSize; ++i)
    {
        params.peer_barrier_ptrs_out[i] = static_buffers->peer_barrier_ptrs_out[i];
    }
    // Assume that a single allreduce will not be divided into more than 64 allreduces of 64MB each,it is not very safe
    params.barrier_flag = flag_value;
    params.ranks_per_node = tpSize;
    params.local_rank = tpRank;

    params.numa_rank = tpRank % LP_ALLREDUCE_MAX_RANKS_PER_NUMA;

    // assume quant_type is 1 bytes , so we can transfer LP_ALLREDUCE_BYTES_PER_LOAD elts once
    int REAL_ELTS_PER_BLOCK
        = (LP_ALLREDUCE_WARPSIZE - 1) * LP_ALLREDUCE_BYTES_PER_LOAD * LP_ALLREDUCE_WARP_NUM_PER_BLOCK;
    int QUANT_ELTS_PER_BLOCK = LP_ALLREDUCE_DEFAULT_BLOCK_SIZE * LP_ALLREDUCE_BYTES_PER_LOAD;

    int max_rounds = (((LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE - 1) / REAL_ELTS_PER_BLOCK + 1) - 1)
            / LP_ALLREDUCE_MAX_RANKS_PER_NUMA / LP_ALLREDUCE_MAX_BLOCKS
        + 1;
    int max_fence_rounds = (max_rounds - 1) / LP_ALLREDUCE_BUFFER_DUPLICATE + 1;

    uint64_t quantize_offset = max_fence_rounds * LP_ALLREDUCE_MAX_RANKS_PER_NUMA * LP_ALLREDUCE_MAX_BLOCKS
        * LP_ALLREDUCE_BUFFER_DUPLICATE * QUANT_ELTS_PER_BLOCK;
    for (int i = 0; i < LP_ALLREDUCE_MAX_RANKS_PER_NUMA; ++i)
    {
        params.inputs_inside_numa[i]
            = params.peer_comm_buffer_ptrs[(tpRank / LP_ALLREDUCE_MAX_RANKS_PER_NUMA) * LP_ALLREDUCE_MAX_RANKS_PER_NUMA
                + i];
    }

    for (int i = 0; i < LP_ALLREDUCE_MAX_BLOCKS; ++i)
    {

        const size_t block_buffer_size
            = QUANT_ELTS_PER_BLOCK * LP_ALLREDUCE_BUFFER_CHUNKS * LP_ALLREDUCE_BUFFER_DUPLICATE;
        char* base_ptr = reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank]);

        params.rs_buffers[i] = base_ptr + quantize_offset + block_buffer_size * i;

        const size_t ar_buffer_offset = quantize_offset + block_buffer_size * LP_ALLREDUCE_MAX_BLOCKS;

        params.ar_buffers[i] = base_ptr + ar_buffer_offset + block_buffer_size * i;

        int const cross_numa_rank = (tpRank + LP_ALLREDUCE_MAX_RANKS_PER_NUMA) % tpSize;
        params.ar_peer_buffers_cross_numa[i] = reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[cross_numa_rank])
            + quantize_offset + block_buffer_size * i;
        int const numa_group_base = (tpRank / LP_ALLREDUCE_MAX_RANKS_PER_NUMA) * LP_ALLREDUCE_MAX_RANKS_PER_NUMA;
        for (int j = 0; j < LP_ALLREDUCE_MAX_RANKS_PER_NUMA; ++j)
        {
            int const rank_in_numa = numa_group_base + j;
            params.ag_peer_buffers_inside_numa[i * LP_ALLREDUCE_MAX_RANKS_PER_NUMA + j]
                = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[rank_in_numa])
                    + ar_buffer_offset + block_buffer_size * i);
        }

        const size_t rs_send_flags_offset = ar_buffer_offset + block_buffer_size * LP_ALLREDUCE_MAX_BLOCKS;
        params.rs_send_flags[i] = reinterpret_cast<uint64_t*>(base_ptr + rs_send_flags_offset + i * sizeof(uint64_t));

        uint64_t rs_ack_flags_offset = rs_send_flags_offset + LP_ALLREDUCE_MAX_BLOCKS * sizeof(uint64_t);
        params.rs_ack_flags[i]
            = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank])
                + rs_ack_flags_offset + i * sizeof(uint64_t) * 2);

        uint64_t rs_notify_local_flags_offset = rs_ack_flags_offset + LP_ALLREDUCE_MAX_BLOCKS * sizeof(uint64_t) * 2;
        params.rs_notify_local_flags[i]
            = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank])
                + rs_notify_local_flags_offset + i * sizeof(uint64_t));

        uint64_t rs_notify_remote_flags_offset
            = rs_notify_local_flags_offset + LP_ALLREDUCE_MAX_BLOCKS * sizeof(uint64_t);

        // now only 8gpus can use hier , so %8 is a magic num
        params.rs_notify_remote_flags[i] = reinterpret_cast<uint64_t*>(
            reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[(tpRank + LP_ALLREDUCE_MAX_RANKS_PER_NUMA) % tpSize])
            + rs_notify_remote_flags_offset + i * sizeof(uint64_t));

        // special flag for ar stage
        params.ar_ack_peer_rs_flags[i]
            = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank])
                + rs_notify_remote_flags_offset + i * sizeof(uint64_t));

        // rs stage handshake done

        // for partial ar stage handshake
        uint64_t ar_send_flags_offset = rs_notify_remote_flags_offset + LP_ALLREDUCE_MAX_BLOCKS * sizeof(uint64_t);
        params.ar_send_flags[i]
            = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank])
                + ar_send_flags_offset + i * sizeof(uint64_t));

        // 2 flag in numa,so use fix *2
        // for ar notify , it is rs_ack_flags
        params.ar_notify_rs_local_flags[i]
            = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank])
                + rs_ack_flags_offset + i * sizeof(uint64_t) * 2);
        // now only 8gpus can use hier , so %8 is a magic num
        params.ar_notify_rs_remote_flags[i] = reinterpret_cast<uint64_t*>(
            reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[(tpRank + LP_ALLREDUCE_MAX_RANKS_PER_NUMA) % tpSize])
            + rs_ack_flags_offset + i * sizeof(uint64_t) * 2 + sizeof(uint64_t));

        uint64_t ar_ack_flags_offset = ar_send_flags_offset + LP_ALLREDUCE_MAX_BLOCKS * sizeof(uint64_t);

        params.ar_ack_flags[i] = reinterpret_cast<uint64_t*>(
            reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank]) + ar_ack_flags_offset + i * sizeof(uint64_t));

        uint64_t ar_notify_ag_flags_offset = ar_ack_flags_offset + LP_ALLREDUCE_MAX_BLOCKS * sizeof(uint64_t);
        params.ar_notify_ag_flags[i]
            = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank])
                + ar_notify_ag_flags_offset + i * sizeof(uint64_t));

        // partial ar stage done

        // for ag stage
        uint64_t ag_send_flags_offset = ar_notify_ag_flags_offset + LP_ALLREDUCE_MAX_BLOCKS * sizeof(uint64_t);
        params.ag_send_flags[i]
            = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank])
                + ag_send_flags_offset + i * sizeof(uint64_t));

        // 4 flag in numa,so use fix *4
        uint64_t ag_ack_peer_inside_numa_flags_offset
            = ag_send_flags_offset + LP_ALLREDUCE_MAX_BLOCKS * sizeof(uint64_t);
        params.ag_ack_peer_inside_numa_flags[i]
            = reinterpret_cast<uint64_t*>(reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[tpRank])
                + ag_ack_peer_inside_numa_flags_offset + i * sizeof(uint64_t) * 4);

        for (int j = 0; j < LP_ALLREDUCE_MAX_RANKS_PER_NUMA; ++j)
        {
            params.ag_notify_peer_inside_numa_flags[i * LP_ALLREDUCE_MAX_RANKS_PER_NUMA + j]
                = reinterpret_cast<uint64_t*>(
                    reinterpret_cast<char*>(params.peer_comm_buffer_ptrs[(tpRank / LP_ALLREDUCE_MAX_RANKS_PER_NUMA)
                            * LP_ALLREDUCE_MAX_RANKS_PER_NUMA
                        + j])
                    + ag_ack_peer_inside_numa_flags_offset + i * sizeof(uint64_t) * 4
                    + (tpRank % LP_ALLREDUCE_MAX_RANKS_PER_NUMA) * sizeof(uint64_t));
        }
        // ag stage done
    }

    return params;
}

bool lowPrecisionConfigurationSupported(size_t n_ranks, size_t msg_size)
{
    size_t elts_per_thread = LP_ALLREDUCE_BYTES_PER_LOAD; // assume quant_type size is 1 bytes
    int msg_align = elts_per_thread;
    if (n_ranks <= 4)
    {
        msg_align *= n_ranks;
    }
    return msg_size % msg_align == 0;
}

int32_t max_workspace_size_lowprecision(int32_t tp_size)
{
    // assume quant_type is 1 byte , so we can transfer LP_ALLREDUCE_BYTES_PER_LOAD elts once
    constexpr int32_t REAL_ELTS_PER_BLOCK
        = (LP_ALLREDUCE_WARPSIZE - 1) * LP_ALLREDUCE_BYTES_PER_LOAD * LP_ALLREDUCE_WARP_NUM_PER_BLOCK;
    constexpr int32_t QUANT_ELTS_PER_BLOCK = LP_ALLREDUCE_DEFAULT_BLOCK_SIZE * LP_ALLREDUCE_BYTES_PER_LOAD;

    int32_t buffer_bytes;
    if (tp_size == 8)
    {
        int32_t max_rounds = ((((LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE - 1) / REAL_ELTS_PER_BLOCK + 1) - 1)
                                 / LP_ALLREDUCE_MAX_RANKS_PER_NUMA / LP_ALLREDUCE_MAX_BLOCKS)
            + 1;
        int32_t max_fence_rounds = ((max_rounds - 1) / LP_ALLREDUCE_BUFFER_DUPLICATE) + 1;
        int32_t quantize_buffer_bytes = max_fence_rounds * LP_ALLREDUCE_MAX_RANKS_PER_NUMA * LP_ALLREDUCE_MAX_BLOCKS
            * LP_ALLREDUCE_BUFFER_DUPLICATE * QUANT_ELTS_PER_BLOCK;
        int32_t comm_buffer_bytes = LP_ALLREDUCE_BUFFER_CHUNKS * LP_ALLREDUCE_BUFFER_DUPLICATE * LP_ALLREDUCE_MAX_BLOCKS
            * LP_ALLREDUCE_HIER_STAGE_NUM * QUANT_ELTS_PER_BLOCK;
        buffer_bytes = quantize_buffer_bytes + comm_buffer_bytes;
    }
    else
    {
        buffer_bytes = (((LP_ALLREDUCE_MAX_ELTS_IN_WORKSPACE / tp_size - 1) / REAL_ELTS_PER_BLOCK) + 1)
            * QUANT_ELTS_PER_BLOCK * tp_size;
    }

    constexpr int32_t HANDSHAKE_FLAG_NUM = 32;
    int32_t flag_bytes = LP_ALLREDUCE_MAX_BLOCKS * HANDSHAKE_FLAG_NUM * sizeof(uint64_t);

    return buffer_bytes + flag_bytes;
}

void customLowPrecisionAllReduce(
    kernels::LowPrecisionAllReduceParams& params, nvinfer1::DataType dataType, hipStream_t stream)
{
    TLLM_CHECK_WITH_INFO(lowPrecisionConfigurationSupported(params.ranks_per_node, params.elts_total),
        "Low Precision Custom all-reduce configuration unsupported");

    sync_check_cuda_error(stream);

    switch (dataType)
    {
    case nvinfer1::DataType::kFLOAT: lowPrecisionAllReduceDispatchType<float>(params, stream); break;
    case nvinfer1::DataType::kHALF: lowPrecisionAllReduceDispatchType<half>(params, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: lowPrecisionAllReduceDispatchType<__hip_bfloat16>(params, stream); break;
#endif
    default: TLLM_THROW("Unsupported dataType for customAllReduce");
    }
    sync_check_cuda_error(stream);
}

} // namespace tensorrt_llm::kernels
