#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2025, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/kernels/sparseAttentionKernels.h"
#include <hipcub/hipcub.hpp>

namespace tensorrt_llm
{
namespace kernels
{
template <int THREADS_PER_BLOCK>
__global__ void gatherKvPageOffsetsKernel(
    int32_t* output_kv_page_offsets, // [num_head_kv, batch_size, 2, max_num_pages_per_seq]
    int32_t* output_seq_lengths,     // [num_head_kv, batch_size]
    int32_t const* kv_page_offsets,  // [batch_size, 2, max_num_pages_per_seq]
    int32_t const* seq_lengths,      // [batch_size]
    SparseAttentionParams const sparse_params, int32_t const batch_size, int32_t const tokens_per_page,
    int32_t const max_num_pages_per_seq)
{
    // Each CUDA block processes one sequence from the batch for one head.
    int32_t const head_idx = blockIdx.x;
    int32_t const batch_idx = blockIdx.y;
    if (batch_idx >= batch_size)
    {
        return;
    }

    // Shared memory for reduction.
    __shared__ typename hipcub::BlockReduce<Pair, THREADS_PER_BLOCK>::TempStorage temp_storage;

    // Get the range of sparse indices and the sequence length.
    int32_t const start_offset = sparse_params.sparse_attn_offsets[batch_idx];
    int32_t const end_offset = sparse_params.sparse_attn_offsets[batch_idx + 1];
    int32_t const total_pages = sparse_params.sparse_attn_offsets[batch_size];
    int32_t const num_sparse_pages = end_offset - start_offset;
    int32_t const original_seq_len = seq_lengths[batch_idx];

    // Get global sparse index.
    int32_t const sparse_idx_global = head_idx * total_pages + start_offset;

    // Get the base memory offset. shape: [batch_size, 2, max_num_pages_per_seq]
    size_t const src_base_offset = (size_t) batch_idx * 2 * max_num_pages_per_seq;
    size_t const dst_base_offset = (size_t) head_idx * batch_size * 2 * max_num_pages_per_seq + src_base_offset;

    // Initialize the local max page index and number of valid pages.
    int32_t local_max_page_index = -1;
    int32_t local_num_valid_pages = 0;

    // Perform the gather operation.
    for (int32_t i = threadIdx.x; i < num_sparse_pages; i += blockDim.x)
    {
        // Get the source idx and offset.
        int32_t const src_idx = sparse_params.sparse_attn_indices[sparse_idx_global + i];
        if (src_idx < 0)
        {
            continue;
        }

        // Update the local max page index.
        local_max_page_index = max(local_max_page_index, src_idx);
        local_num_valid_pages++;

        // Get the source and destination offsets.
        size_t const src_offset_dim0 = src_base_offset + 0 * max_num_pages_per_seq + src_idx;
        size_t const src_offset_dim1 = src_base_offset + 1 * max_num_pages_per_seq + src_idx;
        size_t const dst_offset_dim0 = dst_base_offset + 0 * max_num_pages_per_seq + i;
        size_t const dst_offset_dim1 = dst_base_offset + 1 * max_num_pages_per_seq + i;

        // Perform the gather operation: read from the sparse location and write to the dense location.
        output_kv_page_offsets[dst_offset_dim0] = kv_page_offsets[src_offset_dim0];
        output_kv_page_offsets[dst_offset_dim1] = kv_page_offsets[src_offset_dim1];
    }

    // Reduce the local max page indices and number of valid pages.
    Pair local_pair = {local_max_page_index, local_num_valid_pages};
    Pair result = hipcub::BlockReduce<Pair, THREADS_PER_BLOCK>(temp_storage).Reduce(local_pair, PairReduceOp());

    // Update sequence length for this head and batch.
    if (threadIdx.x == 0)
    {
        int32_t const max_page_index = result.max_val;
        int32_t const num_valid_pages = result.sum_val;
        int32_t const ori_valid_pages = (original_seq_len + tokens_per_page - 1) / tokens_per_page;
        size_t const seq_len_offset = (size_t) head_idx * batch_size + batch_idx;
        if (num_valid_pages > 0)
        {
            int32_t seq_len = original_seq_len - (ori_valid_pages - num_valid_pages) * tokens_per_page;
            int32_t seq_len_remain = original_seq_len % tokens_per_page;
            if (max_page_index != ori_valid_pages - 1 && seq_len_remain != 0)
            {
                seq_len += tokens_per_page - seq_len_remain;
            }
            output_seq_lengths[seq_len_offset] = seq_len;
        }
        else
        {
            output_seq_lengths[seq_len_offset] = 0;
        }
    }
}

// Host-side launcher function
void invokeGatherKvPageOffsets(int32_t* output_kv_page_offsets, int32_t* output_seq_lengths,
    int32_t const* kv_page_offsets, int32_t const* seq_lengths, SparseAttentionParams const sparse_params,
    int32_t const batch_size, int32_t const num_head_kv, int32_t const tokens_per_page,
    int32_t const max_num_pages_per_seq, hipStream_t stream)
{
    // The grid.
    dim3 grid(num_head_kv, batch_size, 1);
    // The block.
    dim3 block(256, 1, 1);
    // Shared memory size.
    size_t smem_size = sizeof(Pair) * 256;

    // Launch the kernel.
    gatherKvPageOffsetsKernel<256><<<grid, block, smem_size, stream>>>(output_kv_page_offsets, output_seq_lengths,
        kv_page_offsets, seq_lengths, sparse_params, batch_size, tokens_per_page, max_num_pages_per_seq);
}
} // namespace kernels
} // namespace tensorrt_llm
