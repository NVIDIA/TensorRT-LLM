#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaBf16Wrapper.h"
#include "tensorrt_llm/common/cudaFp8Utils.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/mathUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/attentionMask.h"
#include "tensorrt_llm/kernels/decoderMaskedMultiheadAttentionUtils.h"
#include "tensorrt_llm/kernels/gptKernels.h"
#include <hipcub/hipcub.hpp>

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{

// A stateful callback functor that maintains the running sum between consecutive scans.
struct BlockPrefixCallbackOp
{
    // Running prefix
    int mRunningTotal;

    // Constructor
    __device__ BlockPrefixCallbackOp(int runningTotal)
        : mRunningTotal(runningTotal)
    {
    }

    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ int operator()(int blockAggregate)
    {
        int oldPrefix = mRunningTotal;
        mRunningTotal += blockAggregate;
        return oldPrefix;
    }
};

// Given an array of sequence lengths, with batchSize elements, that kernel computes the exclusive
// prefix-sums of the sequence lengths. There are (batchSize+1) elements in seqOffsets.
//
// seqOffsets[ 0]        = 0
// seqOffsets[ii]        = seqLengths[0] + .. + seqLengths[ii-1],
// seqOffsets[batchSize] = seqLengths[0] + .. + seqLengths[batchSize-1]
//
// This kernel uses a single thread block of THREADS_PER_BLOCK threads.

// This kernel also computes the padding offsets: Given the index (idx) of a token in a ragged tensor,
// we need the index of the token in the corresponding tensor with padding. We compute an array
// of numTokens elements, called the paddingOffsets, such that the position in the padded tensor
// of the token "idx" in the ragged tensor is given by idx + paddingOffset[idx].
//
// That kernel uses a grid of batchSize blocks.

template <typename T, int THREADS_PER_BLOCK>
__global__ __launch_bounds__(THREADS_PER_BLOCK) void computeSeqAndPaddingOffsets(BuildDecoderInfoParams<T> params)
{
    // Dynamic shared memory for storing seqOffsets.
    extern __shared__ int smem[];
    int* smemSeqQOffsets = (int*) (smem);

    // Fixed Q sequence lengths.
    bool const fixed_q_seqlen = params.seqQLengths == nullptr;

    // Whether to calculate cumulative KV sequence lengths.
    bool const calculate_kv_offsets = params.seqKVOffsets != nullptr;

    // Whether to calculate cumulative packed mask rows.
    bool const calculate_packed_mask_row_offsets = params.packedMaskRowOffsets != nullptr;

    // Whether to calculate cumulative cp partial sequence lengths.
    int const cpSize = params.cpSize;
    bool const calculate_cp_offsets = cpSize > 1 && params.seqCpPartialOffsets != nullptr;

    // Compute the padding offsets for Encoder Inputs.
    bool const need_encoder_padding_offsets = (params.encoderPaddingOffsets != nullptr) && calculate_kv_offsets;
    [[maybe_unused]] int* smemEncoderSeqQOffsets;

    // The implementation of the parallel scan in the thread block (see CUB for details).
    using BlockScan = hipcub::BlockScan<int, THREADS_PER_BLOCK>;

    // Allocate storage in shared memory to do the scan.
    __shared__ typename BlockScan::TempStorage tempQStorage;
    [[maybe_unused]] __shared__ typename BlockScan::TempStorage tempMaskStorage;
    [[maybe_unused]] __shared__ typename BlockScan::TempStorage tempKVStorage;

    // This prefixOp operator keeps a running sum for when we need multiple iterations of the loop.
    BlockPrefixCallbackOp prefixQOp(0);
    BlockPrefixCallbackOp prefixMaskOp(0);
    BlockPrefixCallbackOp prefixKVOp(0);
    BlockPrefixCallbackOp prefixCpPartialOp(0);

    if (need_encoder_padding_offsets)
    {
        smemEncoderSeqQOffsets = (int*) (&smemSeqQOffsets[params.batchSize + 1]);
    }

    // Iterate over the sequences in the batch.
    //
    // The loop index does not depend on the thread index to make sure all the threads enter the
    // loop as we have __syncthreads in it (and we need all threads to participate to avoid
    // deadlocks).
    // Only the last block computes the full sequence offsets.
    bool const storeSeqOffsets = blockIdx.x == (params.batchSize - 1);
    int const batchSizeBound = blockIdx.x + 1;
    for (int batchOffset = 0; batchOffset <= batchSizeBound; batchOffset += THREADS_PER_BLOCK)
    {
        // The index of the batch.
        int batchIdx = batchOffset + threadIdx.x;

        // Threads that correspond to valid sequences read the length.
        int seqQLength = 0;
        [[maybe_unused]] int packedMaskRows = 0;
        [[maybe_unused]] int seqKVLength = 0;
        [[maybe_unused]] int seqCpPartialLength = 0;
        if (batchIdx < batchSizeBound)
        {
            seqQLength = fixed_q_seqlen ? params.maxQSeqLength : params.seqQLengths[batchIdx];
            // Need to pad mask rows to multiple of 128 for each sequence in the batch.
            packedMaskRows = calculate_packed_mask_row_offsets
                ? divUp(seqQLength, int(FLASH_ATTEN_PACKED_MASK_M_ALIGNMENT)) * FLASH_ATTEN_PACKED_MASK_M_ALIGNMENT
                : 0;
            seqKVLength = calculate_kv_offsets ? params.seqKVLengths[batchIdx] : 0;
            seqCpPartialLength = calculate_cp_offsets ? (seqQLength + cpSize - 1) / cpSize : 0;
        }

        // Do the prefix-scan (it calls syncthreads internally).
        int seqQOffset;
        [[maybe_unused]] int packedMaskRowOffset;
        [[maybe_unused]] int seqKVOffset;
        [[maybe_unused]] int seqCpPartialOffset;
        BlockScan(tempQStorage).ExclusiveSum(seqQLength, seqQOffset, prefixQOp);
        if (calculate_packed_mask_row_offsets)
        {
            BlockScan(tempMaskStorage).ExclusiveSum(packedMaskRows, packedMaskRowOffset, prefixMaskOp);
        }
        if (calculate_kv_offsets)
        {
            BlockScan(tempKVStorage).ExclusiveSum(seqKVLength, seqKVOffset, prefixKVOp);
        }
        if (calculate_cp_offsets)
        {
            BlockScan(tempKVStorage).ExclusiveSum(seqCpPartialLength, seqCpPartialOffset, prefixCpPartialOp);
        }

        // Store the result to smem.
        if (batchIdx <= batchSizeBound)
        {
            smemSeqQOffsets[batchIdx] = seqQOffset;
            if (need_encoder_padding_offsets)
            {
                smemEncoderSeqQOffsets[batchIdx] = seqKVOffset;
            }
        }

        // Store the result.
        if (batchIdx <= batchSizeBound && storeSeqOffsets)
        {
            params.seqQOffsets[batchIdx] = seqQOffset;
            if (calculate_packed_mask_row_offsets)
            {
                params.packedMaskRowOffsets[batchIdx] = packedMaskRowOffset;
            }
            if (calculate_kv_offsets)
            {
                params.seqKVOffsets[batchIdx] = seqKVOffset;
            }
            if (calculate_cp_offsets)
            {
                params.seqCpPartialOffsets[batchIdx] = seqCpPartialOffset;
            }
        }

        // Make sure the shared memory can be reused for the next iteration of the loop.
        __syncthreads();
    }

    int batchIdx = blockIdx.x;

    // Compute the padding offsets.
    auto compute_padding_offset = [&](int* smem_offset, int maxSeqLength, int* paddingOffsets)
    {
        // Block x dimension is the batch dimension, while threads iterate all tokens in the sequence.
        int seqBegin = smem_offset[batchIdx];
        // The offset to the 1st element of the next sequence.
        int seqEnd = smem_offset[batchIdx + 1];
        // The length of the sequence.
        int seqLength = seqEnd - seqBegin;
        // The number of padded tokens in the previous sequences.
        int paddingOffset = batchIdx * maxSeqLength - seqBegin;

        // Iterate over the tokens to update the number of padded elements.
        for (int tokenIdx = threadIdx.x; tokenIdx < seqLength; tokenIdx += blockDim.x)
        {
            paddingOffsets[seqBegin + tokenIdx] = paddingOffset;
        }
    };

    if (params.paddingOffsets != nullptr)
    {
        compute_padding_offset(smemSeqQOffsets, params.maxQSeqLength, params.paddingOffsets);
    }

    if (need_encoder_padding_offsets)
    {
        compute_padding_offset(smemEncoderSeqQOffsets, params.maxEncoderQSeqLength, params.encoderPaddingOffsets);
    }

    // Compuate tokens Info (batchIdx, tokenIdxInSeq).
    if (params.tokensInfo != nullptr)
    {
        // The begin of the sequence.
        int seqBegin = params.removePadding ? smemSeqQOffsets[batchIdx] : batchIdx * params.maxQSeqLength;
        // The end of the sequence.
        int seqEnd = params.removePadding ? smemSeqQOffsets[batchIdx + 1] : (batchIdx + 1) * params.maxQSeqLength;
        // FIXME(Eagle): the last sequence needs to consider the paddings.
        if (batchIdx == (params.batchSize - 1))
        {
            seqEnd = std::max(params.numTokens, seqEnd);
        }
        // The length of the sequence.
        int seqLength = seqEnd - seqBegin;

        // Iterate over the tokens to update the number of padded elements.
        for (int tokenIdx = threadIdx.x; tokenIdx < seqLength; tokenIdx += blockDim.x)
        {
            params.tokensInfo[seqBegin + tokenIdx] = make_int2(batchIdx, tokenIdx);
        }
    };

    // Each block generates the rotary embedding inv_freq tensor for the corresponding sequence.
    int zid = 2 * threadIdx.x;
    int halfRotaryEmbeddingDim = params.rotaryEmbeddingDim / 2;
    if (params.rotaryEmbeddingDim > 0 && zid < params.rotaryEmbeddingDim)
    {
        mmha::update_rotary_base_n_scale(params.rotaryEmbeddingBase, params.rotaryEmbeddingScale,
            params.rotaryScalingType, params.rotaryEmbeddingDim, params.rotaryEmbeddingMaxPositions,
            params.seqKVLengths[batchIdx]);
        // Recompute the rotary scales when it is dynamic scaling.
        if (params.rotaryScalingType == RotaryScalingType::kDYNAMIC || params.rotaryEmbeddingInvFreqCache == nullptr)
        {
            float const invFreq = params.rotaryEmbeddingScale
                / powf(params.rotaryEmbeddingBase, zid / (float) params.rotaryEmbeddingDim);
            params.rotaryEmbeddingInvFreq[batchIdx * halfRotaryEmbeddingDim + threadIdx.x] = invFreq;
        }
        else
        {
            // Otherwise, expand the inv freq cache to batch size.
            float const invFreqCache = params.rotaryEmbeddingInvFreqCache[threadIdx.x];
            params.rotaryEmbeddingInvFreq[batchIdx * halfRotaryEmbeddingDim + threadIdx.x] = invFreqCache;
        }
    }

    // Prepare values for fmha.
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        // Reset fmha tile counter to 0 before launching fmha kernels.
        if (params.fmhaTileCounter)
        {
            params.fmhaTileCounter[0] = 0u;
        }
        // Take the quantization scales into consideration.
        int const q_scale_idx = 0;
        int const k_scale_idx = params.separateQkvScales ? 1 : 0;
        int const v_scale_idx = params.separateQkvScales ? 2 : 0;
        float dequantScaleQ = params.dequantScaleQkv ? params.dequantScaleQkv[q_scale_idx] : 1.f;
        float dequantScaleK = params.dequantScaleQkv ? params.dequantScaleQkv[k_scale_idx] : 1.f;
        float dequantScaleV = params.dequantScaleQkv ? params.dequantScaleQkv[v_scale_idx] : 1.f;

        float quantScaleO = params.quantScaleO ? params.quantScaleO[0] : 1.f;
        if (params.fmhaBmm1Scale)
        {
            // The scale after fmha bmm1.
            params.fmhaBmm1Scale[0] = dequantScaleQ * dequantScaleK * params.fmhaHostBmm1Scale;
            // The scale prepared for log2 optimization.
            constexpr float kLog2e = 1.4426950408889634074f;
            params.fmhaBmm1Scale[1] = params.fmhaBmm1Scale[0] * kLog2e;
        }
        if (params.fmhaBmm2Scale)
        {
            // The scale after fmha bmm2.
            params.fmhaBmm2Scale[0] = quantScaleO * dequantScaleV;
        }
    }
}

template <typename T>
void invokeBuildDecoderInfo(BuildDecoderInfoParams<T> const& params, hipStream_t stream)
{
    // Compute the sequence and padding offsets.
    int const THREADS_PER_BLOCK = 256;
    TLLM_CHECK_WITH_INFO(params.rotaryEmbeddingDim / 2 <= 256 && params.rotaryEmbeddingDim % 2 == 0,
        "Rotary embedding dim is assumed to be smaller than 512 and multiple of 2.");
    TLLM_CHECK_WITH_INFO(
        !(params.seqKVLengths == nullptr && params.rotaryEmbeddingDim > 0), "KV sequence lengths buffer is invalid.");
    bool const need_encoder_padding_offsets
        = (params.encoderPaddingOffsets != nullptr) && (params.seqKVOffsets != nullptr);
    const size_t smem_size
        = (need_encoder_padding_offsets ? (params.batchSize + 1) * 2 : (params.batchSize + 1)) * sizeof(int);
    computeSeqAndPaddingOffsets<T, THREADS_PER_BLOCK>
        <<<params.batchSize, THREADS_PER_BLOCK, smem_size, stream>>>(params);

    // Compute the attention mask, if needed.
    if (params.attentionMask != nullptr)
    {
        TLLM_CHECK_WITH_INFO(params.seqQLengths != nullptr, "Q sequence lengths buffer is invalid.");
        AttentionMaskParams<T> attentionMaskParams;
        memset((void*) &attentionMaskParams, 0, sizeof(attentionMaskParams));
        // Set parameters.
        attentionMaskParams.mask = params.attentionMask;
        // Nullptr indicates that the row dimension are not packed (i.e. paddings are not removed).
        attentionMaskParams.cuQSeqLens = nullptr;
        attentionMaskParams.actualQSeqLens = params.seqQLengths;
        attentionMaskParams.actualKvSeqLens = params.seqQLengths;
        attentionMaskParams.attentionMaskType = params.attentionMaskType;
        attentionMaskParams.blockSparseParams = params.blockSparseParams;
        attentionMaskParams.batchSize = params.batchSize;
        attentionMaskParams.maxQSeqLen = params.maxQSeqLength;
        attentionMaskParams.maxKvSeqLen = params.maxQSeqLength;
        attentionMaskParams.slidingWindowSize = params.attentionWindowSize;
        // Launch the kernel.
        invokeBuildAttentionMask(attentionMaskParams, stream);
    }
}

template void invokeBuildDecoderInfo(BuildDecoderInfoParams<float> const&, hipStream_t);
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<half> const&, hipStream_t);
#ifdef ENABLE_BF16
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<__hip_bfloat16> const&, hipStream_t);
#endif
#ifdef ENABLE_FP8
template void invokeBuildDecoderInfo(BuildDecoderInfoParams<__hip_fp8_e4m3_fnuz> const&, hipStream_t);
#endif

__global__ void updatePaddingCountKernel(int* paddingPerSeq, int const* seqLengths, int maxQSeqLength, int batchSize)
{

    for (int ii = threadIdx.x; ii < batchSize; ii += blockDim.x)
    {
        paddingPerSeq[ii] = maxQSeqLength - seqLengths[ii];
    }
}

} // namespace kernels
} // namespace tensorrt_llm
