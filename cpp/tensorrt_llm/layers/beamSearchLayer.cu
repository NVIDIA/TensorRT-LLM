#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "beamSearchLayer.h"
#include "tensorrt_llm/kernels/beamSearchKernels/beamSearchKernelsTemplate.h"

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/stringUtils.h"
#include "tensorrt_llm/kernels/beamSearchKernels.h"
#include "tensorrt_llm/layers/defaultDecodingParams.h"
#include "tensorrt_llm/layers/layerUtils.h"
#include <limits>

using namespace tensorrt_llm::runtime;
using namespace tensorrt_llm::kernels;

namespace tensorrt_llm::layers
{

#define GET_INFO_STAGE1(paddedBeamWidth)                                                                               \
    {                                                                                                                  \
        int constexpr nBlock = (paddedBeamWidth < 16) ? ((paddedBeamWidth < 8) ? kThreadForSmallBeamWidth : 128) : 64; \
        TLLM_CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(                                                 \
            &nMaxActiveBlock, beamStage1Kernel<T, 2 * paddedBeamWidth, nBlock>, nBlock, 0));                           \
        TLLM_CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(beamStage1Kernel<T), 2 * paddedBeamWidth, nBlock>));               \
        break;                                                                                                         \
    }

#define GET_INFO_STAGE2(paddedBeamWidth)                                                                               \
    {                                                                                                                  \
        if (nByteDynamicSharedMemoryStage2 > nByteMaxSharedMemoryPerBlock)                                             \
        {                                                                                                              \
            TLLM_CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(beamStage2Kernel<T), paddedBeamWidth, 128, false>));           \
        }                                                                                                              \
        else if (nVPart <= 32)                                                                                         \
        {                                                                                                              \
            TLLM_CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(beamStage2Kernel<T), paddedBeamWidth, 32, true>));             \
        }                                                                                                              \
        else if (nVPart <= 64)                                                                                         \
        {                                                                                                              \
            TLLM_CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(beamStage2Kernel<T), paddedBeamWidth, 64, true>));             \
        }                                                                                                              \
        else                                                                                                           \
        {                                                                                                              \
            TLLM_CUDA_CHECK(hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(beamStage2Kernel<T), paddedBeamWidth, 128, true>));            \
        }                                                                                                              \
        break;                                                                                                         \
    }

#define GET_INFO_STAGE3(paddedBeamWidth, isV2)                                                                         \
    {                                                                                                                  \
        int constexpr nThreadStage3 = (paddedBeamWidth + 31) / 32 * 32;                                                \
        TLLM_CUDA_CHECK(                                                                                               \
            hipFuncGetAttributes(&attr, reinterpret_cast<const void*>(beamStage3Kernel<T), paddedBeamWidth, nThreadStage3, true, isV2>));            \
        break;                                                                                                         \
    }

template <typename T>
BeamSearchLayer<T>::BeamSearchLayer(DecoderDomain const& decoderDomain, std::shared_ptr<BufferManager> bufferManager)
    : BaseLayer(decoderDomain, bufferManager)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    SizeType32 const batchSize{mDecoderDomain.getBatchSize()};
    SizeType32 const beamWidth{mDecoderDomain.getBeamWidth()};
    SizeType32 const vocabSize{mDecoderDomain.getVocabSize()};
    TLLM_CHECK_WITH_INFO(beamWidth <= kMaxBeamWidth, "Beam width is larger than the maximum supported (%d > %d)",
        int(beamWidth), int(kMaxBeamWidth));
    this->mVBWS = decoderDomain.getUseVariableBeamWidthSearch();

    allocateBuffer();
    configureBeamSearchLayer();

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::allocateBuffer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    SizeType32 const batchSize{mDecoderDomain.getBatchSize()};
    auto const batchSizeShape{ITensor::makeShape({batchSize})};
    auto const batchSizeXBeamWidthArraySizeShape{
        ITensor::makeShape({batchSize * static_cast<SizeType32>(kMaxBeamWidthArrayLength)})};

    mBeamSearchDiversityRateHost = mBufferManager->pinnedPool(batchSizeShape, TRTDataType<float>::value);
    mBeamSearchDiversityRateDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<float>::value);

    mLengthPenaltyHost = mBufferManager->pinnedPool(batchSizeShape, TRTDataType<float>::value);
    mLengthPenaltyDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<float>::value);

    mEarlyStoppingHost = mBufferManager->pinnedPool(batchSizeShape, TRTDataType<int>::value);
    mEarlyStoppingDevice = mBufferManager->gpu(batchSizeShape, TRTDataType<int>::value);

    mBeamWidthArrayHost = mBufferManager->pinnedPool(batchSizeXBeamWidthArraySizeShape, TRTDataType<int>::value);
    mBeamWidthArrayDevice = mBufferManager->gpu(batchSizeXBeamWidthArraySizeShape, TRTDataType<int>::value);

    mBeamWidthIn = mBufferManager->pinnedPool(batchSizeShape, TRTDataType<int>::value);
    mBeamWidthOut = mBufferManager->pinnedPool(batchSizeShape, TRTDataType<int>::value);

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::configureBeamSearchLayer()
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    SizeType32 const batchSize{mDecoderDomain.getBatchSize()};
    SizeType32 const beamWidth{mDecoderDomain.getBeamWidth()};
    SizeType32 const vocabSize{mDecoderDomain.getVocabSize()};
    SizeType32 const paddedBeamWidth{padToNextPowerOfTwo(beamWidth)};
    hipFuncAttributes attr;

    // Find device information to determine `nVPart`.
    int const nByteMaxSharedMemoryPerSM = getMaxSharedMemoryPerSM();
    int const nByteMaxSharedMemoryPerBlock = getMaxSharedMemoryPerBlockOptin();
    int const nByteReservedSharedMemoryPerBlock = nByteMaxSharedMemoryPerSM - nByteMaxSharedMemoryPerBlock;
    this->mByteMaxSharedMemoryPerBlock = nByteMaxSharedMemoryPerBlock;

    if (beamWidth <= kMaxBeamWidthForV1 && !(this->mVBWS))
    {
        // V1 workflow for small beam width and non-VBWS
        // Stage 1
        int nMaxActiveBlock = -1;
        switch (paddedBeamWidth)
        {
        case 1: GET_INFO_STAGE1(1);
        case 2: GET_INFO_STAGE1(2);
        case 4: GET_INFO_STAGE1(4);
        case 8: GET_INFO_STAGE1(8);
        default: break;
        }
        int nByteStaticSharedMemory = attr.sharedSizeBytes;
        int nByteMaxDynamicSharedMemoryPerBlock = nByteMaxSharedMemoryPerBlock - nByteStaticSharedMemory;
        // Find the maximum of `nBlock` (maximum of `nVPart`, minimum of `nByteDynamicSharedMemoryStage1`), s.t.
        // `nVPart <= kMaxVPartStage1 && nByteDynamicSharedMemoryStage1 * nVPart >= sizeof(T) * vocabSize`
        TLLM_CHECK_WITH_INFO(nByteMaxDynamicSharedMemoryPerBlock * kMaxVPartStage1 >= sizeof(T) * vocabSize,
            "vocab_size is too large for Beam search.");
        int nByteExtralSharedMemory = nByteReservedSharedMemoryPerBlock + nByteStaticSharedMemory;
        int nBlock = nMaxActiveBlock;
        int nVPart = kMaxVPartStage1 + 1;
        for (; nBlock > 0 && nVPart > kMaxVPartStage1; --nBlock)
        {
            int nByteDynamicSharedMemoryStage1 = nByteMaxSharedMemoryPerSM / nBlock - nByteExtralSharedMemory;
            nByteDynamicSharedMemoryStage1 -= nByteDynamicSharedMemoryStage1 % sizeof(T);
            nVPart = ceilDiv(sizeof(T) * vocabSize, nByteDynamicSharedMemoryStage1);
        }
        TLLM_CHECK_WITH_INFO(nBlock >= 0, "No enough active blocks for Beam Search stage 1 kernel.");

        int const nByteDynamicSharedMemoryStage1 = sizeof(T) * ceilDiv(vocabSize, nVPart);
        this->mVPart = nVPart;
        this->mByteSharedMemoryStage1 = nByteDynamicSharedMemoryStage1; // Only dynamic shared memory

        // Stage 2
        TLLM_CHECK_WITH_INFO(batchSize * beamWidth * paddedBeamWidth < (1 << 21),
            "max_batch_size or max_beam_width of TRT-LLM engine is too large for Beam search, try to decrease the "
            "parameters while building.");
        size_t const nByteDynamicSharedMemoryStage2 = common::roundUp(
            sizeof(float) * nVPart * (paddedBeamWidth * 4) + sizeof(hipcub::KeyValuePair<int, T>) * paddedBeamWidth * 2,
            4);
        switch (paddedBeamWidth)
        {
        case 1: GET_INFO_STAGE2(1);
        case 2: GET_INFO_STAGE2(2);
        case 4: GET_INFO_STAGE2(4);
        case 8: GET_INFO_STAGE2(8);
        default: break;
        }
        nByteStaticSharedMemory = attr.sharedSizeBytes;
        nByteMaxDynamicSharedMemoryPerBlock = nByteMaxSharedMemoryPerBlock - nByteStaticSharedMemory;
        nByteExtralSharedMemory = nByteReservedSharedMemoryPerBlock + nByteStaticSharedMemory;
        bool const bUseGlobalMemoryStage2 = (nByteDynamicSharedMemoryStage2 > nByteMaxDynamicSharedMemoryPerBlock);

        // Stage 3
        // Keep top 2K candidates in case of k candidates finishes in one iteration
        size_t const nByteDynamicSharedMemoryStage3
            = common::roundUp(sizeof(T) * paddedBeamWidth * paddedBeamWidth * 2, 4);
        switch (paddedBeamWidth)
        {
        case 1: GET_INFO_STAGE3(1, false);
        case 2: GET_INFO_STAGE3(2, false);
        case 4: GET_INFO_STAGE3(4, false);
        case 8: GET_INFO_STAGE3(8, false);
        }
        nByteStaticSharedMemory = attr.sharedSizeBytes;
        nByteMaxDynamicSharedMemoryPerBlock = nByteMaxSharedMemoryPerBlock - nByteStaticSharedMemory;
        nByteExtralSharedMemory = nByteReservedSharedMemoryPerBlock + nByteStaticSharedMemory;
        bool const bUseGlobalMemoryStage3 = (nByteDynamicSharedMemoryStage3 > nByteMaxDynamicSharedMemoryPerBlock);
        this->mByteSharedMemoryStage3 = nByteStaticSharedMemory; // Only static shared memory

        // Compute workspace size, see `beamSearchKernelsTemplate.h` for detailed information
        // |<----- Workspace ----->|
        // |<- A ->|<- B ->|<- C ->|
        //         |<---- D ---->|
        // A for data exchange between stage 2 and 3
        // B for data exchange between stage 1 and 2, can be reuse for stage 3
        // C for stage 2 if `bUseGlobalMemoryStage2 == true`, can be reuse for stage 3
        // D for stage 3 if `bUseGlobalMemoryStage3 == true`
        size_t const nByteA = common::roundUp(sizeof(T) * batchSize * paddedBeamWidth * paddedBeamWidth * 4, 4);
        size_t const nByteB
            = common::roundUp(sizeof(T) * batchSize * paddedBeamWidth * kMaxVPartStage1 * paddedBeamWidth * 4, 4);
        size_t const nByteC = (bUseGlobalMemoryStage2) ? nByteDynamicSharedMemoryStage2 : 0;
        size_t const nByteD = (bUseGlobalMemoryStage3) ? nByteDynamicSharedMemoryStage3 : 0;
        this->mWorkspaceSize = nByteA + std::max(nByteB + nByteC, nByteD);
    }
    else // V2 workflow for large beam width or VBWS
    {
        this->mV2 = true;
        switch (paddedBeamWidth)
        {
        case 1: GET_INFO_STAGE3(1, true);
        case 2: GET_INFO_STAGE3(2, true);
        case 4: GET_INFO_STAGE3(4, true);
        case 8: GET_INFO_STAGE3(8, true);
        case 16: GET_INFO_STAGE3(16, true);
        case 32: GET_INFO_STAGE3(32, true);
        case 64: GET_INFO_STAGE3(64, true);
        case 128: GET_INFO_STAGE3(128, true);
        case 256: GET_INFO_STAGE3(256, true);
        case 512: GET_INFO_STAGE3(512, true);
        case 1024: GET_INFO_STAGE3(1024, true);
        }
        this->mByteSharedMemoryStage3 = attr.sharedSizeBytes; // Only static shared memory

        // Compute shared memory size for stage 3
        // Compute workspace size, see `beamSearchKernelsTemplate.h` for detailed information
        // |<----------------------------------------- Workspace ------------------------------------------>|
        // |<- Stage2Ids ->|<- Stage2LogProbs ->|<- Stage1Ids ->|<- Stage1LogProbs ->|<---- Stage1TopK ---->|
        //                                                                           |<- stage2TopK ->|
        //                                      |<------------------ Stage3 ------------------>|
        SizeType32 const batchSize{mDecoderDomain.getBatchSize()};
        SizeType32 const beamWidth{mDecoderDomain.getBeamWidth()};
        SizeType32 const vocabSize{mDecoderDomain.getVocabSize()};
        SizeType32 const paddedBeamWidth{padToNextPowerOfTwo(beamWidth)};
        size_t const nByteStage1LogProbs = roundUp(sizeof(T) * batchSize * paddedBeamWidth * paddedBeamWidth * 2, 4);
        size_t const nByteStage1Ids = roundUp(sizeof(int) * batchSize * paddedBeamWidth * paddedBeamWidth * 2, 4);
        size_t const nByteStage2LogProbs = roundUp(sizeof(T) * batchSize * paddedBeamWidth * 2, 4);
        size_t const nByteStage2Ids = roundUp(sizeof(int) * batchSize * paddedBeamWidth * 2, 4);
        size_t const nByteStage1TopK
            = invokeComputeTopkLastDimWorkspaceSize<T>(batchSize * beamWidth, vocabSize, paddedBeamWidth * 2, true);
        size_t const nByteStage2TopK = invokeComputeTopkLastDimWorkspaceSize<T>(
            batchSize, paddedBeamWidth * paddedBeamWidth * 2, beamWidth * 2, true);
        size_t const nByteStage3 = sizeof(T) * beamWidth * beamWidth * 2;
        this->mWorkspaceSize = nByteStage2LogProbs + nByteStage2Ids
            + max(nByteStage1LogProbs + nByteStage1Ids + max(nByteStage1TopK, nByteStage2TopK), nByteStage3);
    }

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
size_t BeamSearchLayer<T>::getWorkspaceSize() const noexcept
{
    return mWorkspaceSize;
}

template <typename T>
void BeamSearchLayer<T>::setup(SizeType32 const batchSize, SizeType32 const beamWidth, TensorConstPtr batchSlots,
    std::shared_ptr<BaseSetupParams> const& baseSetupParams,
    std::shared_ptr<runtime::DecodingLayerWorkspace> const& workspace)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    SizeType32 const maxBamWidth{mDecoderDomain.getBeamWidth()};
    TLLM_CHECK_WITH_INFO(beamWidth <= maxBamWidth, "Beam width is larger than the constructed for (%d > %d).",
        int(beamWidth), int(maxBamWidth));

    auto setupParams = std::dynamic_pointer_cast<BeamSearchSetupParams>(baseSetupParams);
    auto constexpr fltMax = std::numeric_limits<float>::max();
    auto constexpr fltMin = std::numeric_limits<float>::lowest();
    auto constexpr fltEpsilon = std::numeric_limits<float>::epsilon();
    auto constexpr int32Max = std::numeric_limits<int32_t>::max();
    FillBuffers const fillBuffers{batchSize, mDecoderDomain.getBatchSize(), mBufferManager};
    fillBuffers(setupParams->beamSearchDiversityRate, DefaultDecodingParams::getBeamSearchDiversity(),
        mBeamSearchDiversityRateHost, mBeamSearchDiversityRateDevice, batchSlots, std::make_pair(-fltEpsilon, fltMax),
        "diversity rate");
    fillBuffers(setupParams->lengthPenalty, DefaultDecodingParams::getLengthPenalty(), mLengthPenaltyHost,
        mLengthPenaltyDevice, batchSlots, std::make_pair(fltMin, fltMax), "length penalty");
    fillBuffers(setupParams->earlyStopping, DefaultDecodingParams::getEarlyStopping(), mEarlyStoppingHost,
        mEarlyStoppingDevice, batchSlots, std::make_pair(-fltEpsilon, int32Max), "early stopping");
    fillBuffers(setupParams->beamWidthArray, DefaultDecodingParams::getBeamWidthArray(), mBeamWidthArrayHost,
        mBeamWidthArrayDevice, batchSlots, std::make_pair(-fltEpsilon, kMaxBeamWidth), "beam width array");

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template <typename T>
void BeamSearchLayer<T>::forwardAsync(std::shared_ptr<BaseDecodingOutputs> const& baseOutputs,
    std::shared_ptr<BaseDecodingInputs> const& baseInputs,
    std::shared_ptr<runtime::DecodingLayerWorkspace> const& workspace)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);

    auto ip = std::dynamic_pointer_cast<DecodingInputs>(baseInputs);
    auto op = std::dynamic_pointer_cast<BeamSearchOutputs>(baseOutputs);
    auto const localDecoderDomain = getLocalDecoderDomain(ip, mDecoderDomain);

    TLLM_CHECK_WITH_INFO(localDecoderDomain.getBeamWidth() > 1, "Use beamWidth <= 1 (%d <= 1) in Beam Search mode",
        localDecoderDomain.getBeamWidth());
    TLLM_CHECK_WITH_INFO(ip->srcCacheIndirection.has_value(), "srcCacheIndirection is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(op->parentIds.has_value(), "parentIds tensor is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(op->finished.has_value(), "finished tensor is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(op->cumLogProbs.has_value(), "cumLogProbs tensor is mandatory in beam search.");
    TLLM_CHECK_WITH_INFO(op->beamHypotheses, "Output BeamHypotheses is not set.");
    TLLM_CHECK_WITH_INFO(bufferCastOrNull<int>(*op->sequenceLength) != nullptr || mLengthPenaltyDevice == nullptr,
        "Current sequence lengths must be set for length penalty computation.");
    TLLM_CHECK_WITH_INFO(ip->ite == 0, "Pipeline Parallelism is not supported yet!");

    BeamHypotheses bh;
    // bh's members not used in this function: outputIds, logProbs, outputIdsUnfinish, parentIdsUnfinish
    bh.bVBWS = this->mVBWS;
    bh.nMaxBatchSize = static_cast<std::int32_t>(op->outputIdsPtr->getDimension<0>());
    bh.nBatchSize = ip->localBatchSize;
    bh.nBeamWidth = op->outputIds->getDimension<1>();
    bh.nMaxSeqLen = op->outputIds->getDimension<2>();
    bh.nVocabSize = mDecoderDomain.getVocabSizePadded();
    bh.nVPart = this->mVPart;
    bh.nByteMaxSharedMemoryPerBlock = this->mByteMaxSharedMemoryPerBlock;
    bh.nByteSharedMemoryStage1 = this->mByteSharedMemoryStage1;
    bh.nByteSharedMemoryStage3 = this->mByteSharedMemoryStage3;

    bh.diversityRates = bufferCast<float>(*mBeamSearchDiversityRateDevice);
    bh.lengthPenalties = bufferCast<float>(*mLengthPenaltyDevice);
    bh.earlyStoppings = bufferCast<int>(*mEarlyStoppingDevice);
    bh.beamWidthArraysHost = bufferCast<int>(*mBeamWidthArrayHost);
    bh.beamWidthArraysDevice = bufferCast<int>(*mBeamWidthArrayDevice);

    bh.nBeamWidthInHost = bufferCast<int>(*mBeamWidthIn);
    bh.nBeamWidthOutHost = bufferCast<int>(*mBeamWidthOut);
    if (this->mVBWS)
    {
        int const* batchSlotsHost = bufferCast<int>(*ip->batchSlots);
        for (int i = 0; i < ip->localBatchSize; ++i)
        {
            int const slot = batchSlotsHost[i];
            int const step = ip->beamSearchSteps.value()[slot];
            // Clamp `step` to [0, kMaxBeamWidthArrayLength - 1], and set `indexInput=0` when step = 0 or 1
            int const indexInput = std::min(std::max((int) step - 1, 0), (int) kMaxBeamWidthArrayLength - 1);
            int const indexOutput = std::min((int) step, (int) kMaxBeamWidthArrayLength - 1);
            bh.nBeamWidthInHost[i] = bh.beamWidthArraysHost[slot * kMaxBeamWidthArrayLength + indexInput];
            bh.nBeamWidthOutHost[i] = bh.beamWidthArraysHost[slot * kMaxBeamWidthArrayLength + indexOutput];
        }
    }

    bh.inputLengths = bufferCast<SizeType32>(*ip->inputLengths.value());
    bh.endIds = bufferCast<TokenIdType>(*ip->endIds);
    bh.batchSlots = workspace->getDeviceBatchSlotsPtr(); // Device copy of `ip->batchSlots`

    bh.logProbsTiled = bufferCastOrNull<float>(op->outputLogProbsTiled);
    bh.sequenceLengths = bufferCast<SizeType32>(*op->sequenceLength.value());
    bh.cumLogProbs = bufferCast<float>(*op->cumLogProbs.value());

    bh.outputIdsCBA = op->beamHypotheses->outputIdsCBA;
    bh.logProbsCBA = op->beamHypotheses->logProbsCBA;
    bh.sequenceLengthsCBA = op->beamHypotheses->sequenceLengthsCBA;
    bh.cumLogProbsCBA = op->beamHypotheses->cumLogProbsCBA;
    bh.normedScoresCBA = op->beamHypotheses->normedScoresCBA;
    bh.numBeamsCBA = op->beamHypotheses->numBeamsCBA;
    bh.minNormedScoresCBA = op->beamHypotheses->minNormedScoresCBA;

    bh.batchDones = op->beamHypotheses->batchDones;
    bh.finished = reinterpret_cast<FinishedState*>(bufferCast<FinishedState::UnderlyingType>(*op->finished.value()));

    bh.outputIdsPtr = bufferCast<TokenIdType*>(*op->outputIdsPtr);
    bh.parentIdsPtr = bufferCast<TokenIdType*>(*op->parentIdsPtr);

    T const* logProbs = bufferCast<T>(*workspace->getDeviceRuntimeLogits());
    T const* bias = static_cast<T const*>(nullptr);
    TLLM_CHECK_WITH_INFO(getWorkspaceSize() >= 2 * bh.nBatchSize * bh.nBeamWidth * bh.nBeamWidth * 2,
        "Workspace size (%lu) is not enough for topk softmax required (%lu).", (uint64_t) getWorkspaceSize(),
        (uint64_t) (2 * bh.nMaxBatchSize * bh.nBeamWidth * bh.nBeamWidth * 2));

    if (this->mV2 || this->mVBWS)
    {
        invokeTopkBeamSearch<T, true>(logProbs, bias, workspace->getRawWorkspaceDevicePtr(), bh, getStream());
    }
    else
    {
        invokeTopkBeamSearch<T, false>(logProbs, bias, workspace->getRawWorkspaceDevicePtr(), bh, getStream());
    }

    int* tgtCI = bufferCast<int>(*op->tgtCacheIndirection);
    int* srcCI = bufferCast<int>(*ip->srcCacheIndirection.value());
    invokeUpdateCacheIndirection(tgtCI, srcCI, bh, ip->maxAttentionWindow, ip->sinkTokenLength, getStream());

    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

template class BeamSearchLayer<float>;
template class BeamSearchLayer<half>;

} // namespace tensorrt_llm::layers
