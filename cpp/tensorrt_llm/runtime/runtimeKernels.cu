#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION &
 * AFFILIATES. All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/speculativeDecoding/kvCacheUpdateKernels.h"
#include "tensorrt_llm/runtime/runtimeKernels.h"

#include <NvInferRuntimeBase.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

using namespace tensorrt_llm::runtime;
namespace tc = tensorrt_llm::common;

namespace tensorrt_llm::runtime::kernels
{

namespace
{

template <typename T>
__global__ void fill(T* data, std::size_t size, T const value)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] = value;
    }
}
} // namespace

template <typename T>
void invokeFill(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    fill<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

// template instantiation
template void invokeFill(IBuffer&, SizeType64, CudaStream const&);
template void invokeFill(IBuffer&, std::int32_t, CudaStream const&);
template void invokeFill(IBuffer&, std::int8_t, CudaStream const&);
template void invokeFill(IBuffer&, std::uint8_t, CudaStream const&);
template void invokeFill(IBuffer&, bool, CudaStream const&);
template void invokeFill(IBuffer&, half, CudaStream const&);
template void invokeFill(IBuffer&, float, CudaStream const&);
#ifdef ENABLE_BF16
template void invokeFill(IBuffer&, __hip_bfloat16, CudaStream const&);
#endif // ENABLE_BF16

namespace
{
//! @param data    expected shape [indicesRange, size]
//! @param indices expected shape [gridDim.y]
//! @param size
//! @param values  expected shape [gridDim.y]
template <typename T>
__global__ void fillBatch(T* data, std::int32_t const* indices, std::size_t size, T const* values)
{
    auto const batchIdx = indices[blockIdx.y];
    const T value = values[blockIdx.y];
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const startIdx = batchIdx * size;
    auto const endIdx = startIdx + size;

    for (auto idx = startIdx + tidx; idx < endIdx; idx += stride)
    {
        data[idx] = value;
    }
}

template <typename T>
void invokeFillBatch(IBuffer& buffer, IBuffer const& slotIndices, std::size_t slotStride, IBuffer const& values,
    CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const* const indices = bufferCast<std::int32_t>(slotIndices);
    auto fillValues = bufferCast<T>(values);
    auto numSlots = slotIndices.getSize();
    auto const size = slotStride;
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};

    fillBatch<<<gridSize, blockSize, 0, stream.get()>>>(data, indices, size, fillValues);
}
} // namespace

void invokeFillBatch(IBuffer& buffer, IBuffer const& slotIndices, std::size_t slotStride, IBuffer const& values,
    CudaStream const& stream)
{
    switch (buffer.getDataType())
    {
    case nvinfer1::DataType::kINT32:
        invokeFillBatch<std::int32_t>(buffer, slotIndices, slotStride, values, stream);
        break;
    case nvinfer1::DataType::kINT8:
        invokeFillBatch<std::int8_t>(buffer, slotIndices, slotStride, values, stream);
        break;
    case nvinfer1::DataType::kFLOAT: invokeFillBatch<float>(buffer, slotIndices, slotStride, values, stream); break;
    default: TLLM_THROW("data type not supported");
    }
}

namespace
{
template <typename VecT>
__global__ void copyBatch(uint8_t const* srcData, uint8_t* dstData, SizeType64 const* srcOffsets,
    SizeType64 const* dstOffsets, SizeType64 const* sizes, SizeType64 const dataTypeSize)
{
    constexpr auto VEC_ELTS = static_cast<int32_t>(sizeof(VecT));
    SizeType64 const srcStartIdx = srcOffsets[blockIdx.y] * dataTypeSize;
    SizeType64 const dstStartIdx = dstOffsets[blockIdx.y] * dataTypeSize;
    SizeType64 const size = sizes[blockIdx.y] * dataTypeSize;
    SizeType64 const tidx = (static_cast<SizeType64>(blockIdx.x) * blockDim.x + threadIdx.x) * VEC_ELTS;
    SizeType64 const stride = static_cast<SizeType64>(blockDim.x) * gridDim.x * VEC_ELTS;
    SizeType64 const srcEndIdx = srcStartIdx + size;

    SizeType64 srcIdx = srcStartIdx + tidx;
    SizeType64 dstIdx = dstStartIdx + tidx;

    for (; srcIdx < srcEndIdx; srcIdx += stride, dstIdx += stride)
    {
        *reinterpret_cast<VecT*>(&dstData[dstIdx]) = *reinterpret_cast<VecT const*>(&srcData[srcIdx]);
    }
}
} // namespace

void invokeCopyBatch(IBuffer const& srcBuffer, IBuffer& dstBuffer, IBuffer const& srcOffsets, IBuffer const& dstOffsets,
    IBuffer const& sizes, std::size_t maxStride, CudaStream const& stream)
{
    auto srcDataPtr = reinterpret_cast<uint8_t const*>(srcBuffer.data());
    auto dstDataPtr = reinterpret_cast<uint8_t*>(dstBuffer.data());
    auto srcOffsetsPtr = bufferCast<SizeType64>(srcOffsets);
    auto dstOffsetsPtr = bufferCast<SizeType64>(dstOffsets);
    auto sizesPtr = bufferCast<SizeType64>(sizes);
    auto numSlots = srcOffsets.getSize();
    auto const size = maxStride;
    auto const dataTypeSize = BufferDataType(srcBuffer.getDataType()).getSize();
    auto const copyRowSizeInBytes = size * dataTypeSize;

    auto copyBatchInvocation = copyBatch<uint8_t>;
    auto vectorSize = 1;
    if (dataTypeSize % 16 == 0)
    {
        vectorSize = 16;
        copyBatchInvocation = copyBatch<uint4>;
    }
    else if (dataTypeSize % 8 == 0)
    {
        vectorSize = 8;
        copyBatchInvocation = copyBatch<uint2>;
    }
    else if (dataTypeSize % 4 == 0)
    {
        vectorSize = 4;
        copyBatchInvocation = copyBatch<uint32_t>;
    }
    else if (dataTypeSize % 2 == 0)
    {
        vectorSize = 2;
        copyBatchInvocation = copyBatch<uint16_t>;
    }

    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(copyRowSizeInBytes / vectorSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};
    copyBatchInvocation<<<gridSize, blockSize, 0, stream.get()>>>(
        srcDataPtr, dstDataPtr, srcOffsetsPtr, dstOffsetsPtr, sizesPtr, static_cast<SizeType64>(dataTypeSize));
}

namespace
{
template <typename T>
__global__ void add(T* data, std::size_t size, T const value)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] += value;
    }
}
} // namespace

template <typename T>
void invokeAdd(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    add<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

template void invokeAdd(IBuffer&, std::int32_t, CudaStream const&);
template void invokeAdd(IBuffer&, std::int8_t, CudaStream const&);
template void invokeAdd(IBuffer&, float, CudaStream const&);

namespace
{
template <typename T>
__global__ void reduceSum(T* output, T const* input, std::size_t size)
{
    T threadSum = 0;
    for (auto index = threadIdx.x; index < size; index += blockDim.x)
    {
        threadSum += input[index];
    }

    T blockSum = 0;
    if (blockDim.x <= 32)
    {
        blockSum = tc::warpReduceSum(threadSum);
    }
    else
    {
        blockSum = tc::blockReduceSum(threadSum);
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        *output = blockSum;
    }
}
} // namespace

template <typename T>
void invokeReduce(IBuffer& output, IBuffer const& input, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");
    TLLM_CHECK_WITH_INFO(output.getSize() == 1, common::fmtstr("Output size (%ld) has to be 1", output.getSize()));

    auto outputPtr = bufferCast<T>(output);
    auto inputPtr = bufferCast<T>(input);
    auto const size = input.getSize();

    dim3 blockSize{std::min(512u, static_cast<std::uint32_t>(size))};
    dim3 gridSize{1};

    reduceSum<<<gridSize, blockSize, 0, stream.get()>>>(outputPtr, inputPtr, size);
}

void reduce(IBuffer& output, IBuffer const& input, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeReduce<SizeType32>(output, input, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeReduce<float>(output, input, stream); break;
    case nvinfer1::DataType::kHALF: invokeReduce<half>(output, input, stream); break;
    case nvinfer1::DataType::kINT8: invokeReduce<int8_t>(output, input, stream); break;
    default: TLLM_THROW("data type not supported");
    }
}

namespace
{
__global__ void transpose(
    SizeType32* output, SizeType32 const* input, SizeType32 const batchSize, SizeType32 const rowSize)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType32 tokenIdx = tidx; tokenIdx < rowSize; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * rowSize + tokenIdx;
            auto const outputIdx = tokenIdx * batchSize + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTranspose(ITensor& output, ITensor const& input, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");
    TLLM_CHECK_WITH_INFO(input.getSize() == output.getSize(),
        common::fmtstr("Input size (%ld) and output size (%ld) differ", input.getSize(), output.getSize()));

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));

    SizeType32 const batchSize = inputShape.d[0];
    SizeType32 const rowSize = inputShape.d[1];

    dim3 const blockSize(256, 1);
    dim3 const gridSize((rowSize + blockSize.x - 1) / blockSize.x, batchSize);

    transpose<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<SizeType32>(output), bufferCast<SizeType32 const>(input), batchSize, rowSize);
}

namespace
{
__global__ void transposeWithOutputOffset(SizeType32* output, SizeType32 const* input, SizeType32 const nbInputRows,
    SizeType32 const inputRowSize, SizeType32 const outputRowSize, SizeType32 const outputOffset)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < nbInputRows; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType32 tokenIdx = tidx; tokenIdx < inputRowSize; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * inputRowSize + tokenIdx;
            auto const outputIdx = tokenIdx * outputRowSize + outputOffset + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTransposeWithOutputOffset(
    ITensor& output, ITensor const& input, SizeType32 const outputOffset, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));
    SizeType32 const nbInputRows = inputShape.d[0];
    SizeType32 const inputRowSize = inputShape.d[1];

    auto const& outputShape = output.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 2, common::fmtstr("Output shape must have 2 dimensions, but has %d", outputShape.nbDims));
    SizeType32 const nbOutputRows = outputShape.d[0];
    SizeType32 const outputRowSize = outputShape.d[1];

    TLLM_CHECK_WITH_INFO(inputRowSize == nbOutputRows,
        common::fmtstr("Input dim 1 (%d) and output dim 0 (%d) differ", inputRowSize, nbOutputRows));
    TLLM_CHECK_WITH_INFO(outputOffset + nbInputRows <= outputRowSize,
        common::fmtstr("Input (%d rows) does not fit into output (%d columns, offset %d)", nbInputRows, inputRowSize,
            outputOffset));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((inputRowSize + blockSize.x - 1) / blockSize.x, nbInputRows);

    transposeWithOutputOffset<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType32>(output),
        bufferCast<SizeType32 const>(input), nbInputRows, inputRowSize, outputRowSize, outputOffset);
}

namespace
{
__global__ void transposeWithInputOffset(SizeType32* output, SizeType32 const* input, SizeType32 const outputRowSize,
    SizeType32 const nbOutputRows, SizeType32 const inputRowSize, SizeType32 const inputOffset)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < outputRowSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType32 tokenIdx = tidx; tokenIdx < nbOutputRows; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * inputRowSize + inputOffset + tokenIdx;
            auto const outputIdx = tokenIdx * outputRowSize + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTransposeWithInputOffset(
    ITensor& output, ITensor const& input, SizeType32 const inputOffset, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));
    SizeType32 const nbInputRows = inputShape.d[0];
    SizeType32 const inputRowSize = inputShape.d[1];

    auto const& outputShape = output.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 2, common::fmtstr("Output shape must have 2 dimensions, but has %d", outputShape.nbDims));
    SizeType32 const nbOutputRows = outputShape.d[0];
    SizeType32 const outputRowSize = outputShape.d[1];

    TLLM_CHECK_WITH_INFO(nbInputRows == outputRowSize,
        common::fmtstr("Input dim 0 (%d) and output dim 1 (%d) differ", nbInputRows, outputRowSize));
    TLLM_CHECK_WITH_INFO(inputOffset + nbOutputRows <= inputRowSize,
        common::fmtstr("Cannot extract output (%d rows) from input (%d columns, offset %d)", nbOutputRows, inputRowSize,
            inputOffset));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((nbOutputRows + blockSize.x - 1) / blockSize.x, outputRowSize);

    transposeWithInputOffset<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType32>(output),
        bufferCast<SizeType32 const>(input), outputRowSize, nbOutputRows, inputRowSize, inputOffset);
}

void invokeInclusiveSum(IBuffer& output, IBuffer const& input, BufferManager const& manager, CudaStream const& stream)
{
    auto const size = input.getSize();
    auto const* inputData = bufferCast<SizeType32>(input);
    auto* outputData = bufferCast<SizeType32>(output);

    std::size_t tempStorageBytes{0};
    hipcub::DeviceScan::InclusiveSum(nullptr, tempStorageBytes, inputData, outputData, size, stream.get());
    auto tempStorage = manager.gpu(tempStorageBytes, nvinfer1::DataType::kUINT8);
    auto* tempStorageData = bufferCast<std::uint8_t>(*tempStorage);
    hipcub::DeviceScan::InclusiveSum(tempStorageData, tempStorageBytes, inputData, outputData, size, stream.get());
}

void invokeInclusiveSum(IBuffer& output, IBuffer& tmpBuffer, IBuffer const& input, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(nvinfer1::DataType::kUINT8 == tmpBuffer.getDataType(), "tmpBuffer has wrong data type");

    auto const size = input.getSize();
    auto const* inputData = bufferCast<SizeType32>(input);
    auto* outputData = bufferCast<SizeType32>(output);

    std::size_t tempStorageBytes{0};
    hipcub::DeviceScan::InclusiveSum(nullptr, tempStorageBytes, inputData, outputData, size, stream.get());
    tmpBuffer.resize(tempStorageBytes);
    auto* tmpBufferPtr = bufferCast<std::uint8_t>(tmpBuffer);
    hipcub::DeviceScan::InclusiveSum(tmpBufferPtr, tempStorageBytes, inputData, outputData, size, stream.get());
}

namespace
{
__global__ void buildTokenMask(SizeType32* tokenMask, SizeType32 const* inputLengths, SizeType32 const batchSize,
    SizeType32 const maxInputLength, SizeType32 const maxSeqLength)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType32 tokenIdx = tidx; tokenIdx < maxSeqLength; tokenIdx += blockDim.x * gridDim.x)
        {
            tokenMask[batchIdx * maxSeqLength + tokenIdx]
                = (tokenIdx >= inputLength && tokenIdx < maxInputLength) ? 1 : 0;
        }
    }
}
} // namespace

void invokeBuildTokenMask(
    ITensor& tokenMask, ITensor const& inputLengths, SizeType32 const maxInputLength, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType32>::value == tokenMask.getDataType(), "tokenMask has wrong data type");
    TLLM_CHECK_WITH_INFO(
        TRTDataType<SizeType32>::value == inputLengths.getDataType(), "inputLengths has wrong data type");

    auto const& shape = tokenMask.getShape();
    SizeType32 const batchSize = shape.d[0];
    SizeType32 const maxSeqLength = shape.d[1];

    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "TtokenMask dimension 1 (%d) is smaller than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxSeqLength + blockSize.x - 1) / blockSize.x, batchSize);

    buildTokenMask<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType32>(tokenMask),
        bufferCast<SizeType32 const>(inputLengths), batchSize, maxInputLength, maxSeqLength);
}

namespace
{
__global__ void buildAttentionMask(SizeType32* attentionMask, SizeType32 const size, SizeType32 const padId)
{
    SizeType32 const tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (SizeType32 i = tid; i < size; i += blockDim.x * gridDim.x)
    {
        auto const x = attentionMask[i];
        attentionMask[i] = (x != padId);
    }
}
} // namespace

void invokeBuildAttentionMask(ITensor& attentionMask, SizeType32 const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        TRTDataType<SizeType32>::value == attentionMask.getDataType(), "attentionMask has wrong data type");

    auto const size = attentionMask.getSize();
    dim3 const blockSize(256);
    dim3 const gridSize((size + blockSize.x - 1) / blockSize.x);

    buildAttentionMask<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType32>(attentionMask), size, padId);
}

namespace
{
__global__ void extendAttentionMask(
    SizeType32* newMask, SizeType32 const* oldMask, SizeType32 const batchSize, SizeType32 const seqLength)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType32 tokenIdx = tidx; tokenIdx < seqLength + 1; tokenIdx += blockDim.x * gridDim.x)
        {
            SizeType32 oldIndex = batchIdx * seqLength + tokenIdx;
            SizeType32 newIndex = batchIdx * (seqLength + 1) + tokenIdx;
            newMask[newIndex] = (tokenIdx < seqLength) ? oldMask[oldIndex] : 1;
        }
    }
}
} // namespace

void invokeExtendAttentionMask(ITensor& newMask, ITensor const& oldMask, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType32>::value == newMask.getDataType(), "attentionMask has wrong data type");
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType32>::value == oldMask.getDataType(), "attentionMask has wrong data type");

    auto const& shape = oldMask.getShape();
    SizeType32 const batchSize = shape.d[0];
    SizeType32 const seqLength = shape.d[1];

    dim3 const blockSize(256, 1);
    dim3 const gridSize((seqLength + blockSize.x - 1) / blockSize.x, batchSize);

    extendAttentionMask<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<SizeType32>(newMask), bufferCast<SizeType32>(oldMask), batchSize, seqLength);
}

namespace
{
__global__ void copyInputToOutputTransposed(TokenIdType* outputIds, TokenIdType const* inputIds,
    SizeType32 const* inputLengths, TokenIdType const padId, SizeType32 const batchSize, SizeType32 const beamWidth,
    SizeType32 const maxInputLength)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType32 tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[batchIdx * maxInputLength + tokenIdx] : padId;
            for (SizeType32 beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(tokenIdx, batchIdx, beamIdx, batchSize, beamWidth);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyInputToOutputTransposed(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    TokenIdType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const batchSize = static_cast<SizeType32>(inputLengths.getSize());
    auto const& inputShape = inputIds.getShape();
    SizeType32 const maxInputLength = inputShape.d[inputShape.nbDims - 1];
    auto const& outputShape = outputIds.getShape();
    SizeType32 const maxSeqLength = outputShape.d[0];
    SizeType32 const beamWidth = outputShape.d[2];

    auto const inputBatchSize = inputIds.getSize() / maxInputLength;
    TLLM_CHECK_WITH_INFO(std::size_t(batchSize) == inputBatchSize,
        common::fmtstr("Input ids batch size (%ld) does not match inputLengths size (%ld)", inputBatchSize,
            std::size_t(batchSize)));
    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[1],
        common::fmtstr(
            "Output ids batch size (" FMT_DIM ") does not match inputLengths size (%d)", outputShape.d[1], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyInputToOutputTransposed<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<TokenIdType>(outputIds),
        bufferCast<TokenIdType const>(inputIds), bufferCast<SizeType32 const>(inputLengths), padId, batchSize,
        beamWidth, maxInputLength);
}

namespace
{
__global__ void copyPackedInputToOutputTransposed(TokenIdType* outputIds, TokenIdType const* inputIds,
    SizeType32 const* inputOffsets, TokenIdType const padId, SizeType32 const batchSize, SizeType32 const beamWidth,
    SizeType32 const maxInputLength)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const tokenBegin = inputOffsets[batchIdx];
        auto const tokenEnd = inputOffsets[batchIdx + 1];
        auto const inputLength = tokenEnd - tokenBegin;

        for (SizeType32 tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[tokenBegin + tokenIdx] : padId;
            for (SizeType32 beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(tokenIdx, batchIdx, beamIdx, batchSize, beamWidth);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyPackedInputToOutputTransposed(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputOffsets,
    SizeType32 const maxInputLength, TokenIdType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const batchSize = static_cast<SizeType32>(inputOffsets.getSize()) - 1;
    auto const& outputShape = outputIds.getShape();
    SizeType32 const maxSeqLength = outputShape.d[0];
    SizeType32 const beamWidth = outputShape.d[2];

    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[1],
        common::fmtstr("Output ids batch size (" FMT_DIM ") does not match inputOffsets batch size (%d)",
            outputShape.d[1], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyPackedInputToOutputTransposed<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<TokenIdType>(outputIds),
        bufferCast<TokenIdType const>(inputIds), bufferCast<SizeType32 const>(inputOffsets), padId, batchSize,
        beamWidth, maxInputLength);
}

namespace
{
__global__ void copyInputToOutput(TokenIdType* outputIds, TokenIdType const* inputIds, SizeType32 const* inputLengths,
    TokenIdType const padId, SizeType32 const batchSize, SizeType32 const beamWidth, SizeType32 const maxInputLength,
    SizeType32 const maxSeqLength)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType32 tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[batchIdx * maxInputLength + tokenIdx] : padId;
            for (SizeType32 beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(batchIdx, beamIdx, tokenIdx, beamWidth, maxSeqLength);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyInputToOutput(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    TokenIdType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const& inputShape = inputIds.getShape();
    auto const& outputShape = outputIds.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 3, common::fmtstr("Output shape must have 3 dimensions, but has %d", outputShape.nbDims));

    auto const batchSize = static_cast<SizeType32>(inputLengths.getSize());
    SizeType32 const maxInputLength = inputShape.d[inputShape.nbDims - 1];
    SizeType32 const beamWidth = outputShape.d[1];
    SizeType32 const maxSeqLength = outputShape.d[2];

    auto const inputBatchSize = inputIds.getSize() / maxInputLength;
    TLLM_CHECK_WITH_INFO(std::size_t(batchSize) == inputBatchSize,
        common::fmtstr("Input ids batch size (%ld) does not match inputLengths size (%ld)", inputBatchSize,
            std::size_t(batchSize)));
    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[0],
        common::fmtstr(
            "Output ids batch size (" FMT_DIM ") does not match inputLengths size (%d)", outputShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyInputToOutput<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<TokenIdType>(outputIds),
        bufferCast<TokenIdType const>(inputIds), bufferCast<SizeType32 const>(inputLengths), padId, batchSize,
        beamWidth, maxInputLength, maxSeqLength);
}

namespace
{
__global__ void copyPackedInputToOutput(TokenIdType* outputIds, TokenIdType const* inputIds,
    SizeType32 const* inputOffsets, TokenIdType const padId, SizeType32 const batchSize, SizeType32 const beamWidth,
    SizeType32 const maxInputLength, SizeType32 const maxSeqLength)
{
    SizeType32 const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType32 const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const tokenBegin = inputOffsets[batchIdx];
        auto const tokenEnd = inputOffsets[batchIdx + 1];
        auto const inputLength = tokenEnd - tokenBegin;

        for (SizeType32 tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[tokenBegin + tokenIdx] : padId;
            for (SizeType32 beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(batchIdx, beamIdx, tokenIdx, beamWidth, maxSeqLength);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyPackedInputToOutput(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputOffsets,
    SizeType32 const maxInputLength, TokenIdType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const& outputShape = outputIds.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 3, common::fmtstr("Output shape must have 3 dimensions, but has %d", outputShape.nbDims));

    auto const batchSize = static_cast<SizeType32>(inputOffsets.getSize()) - 1;
    SizeType32 const beamWidth = outputShape.d[1];
    SizeType32 const maxSeqLength = outputShape.d[2];

    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[0],
        common::fmtstr("Output ids batch size (" FMT_DIM ") does not match inputOffsets batch size (%d)",
            outputShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyPackedInputToOutput<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<TokenIdType>(outputIds),
        bufferCast<TokenIdType const>(inputIds), bufferCast<SizeType32 const>(inputOffsets), padId, batchSize,
        beamWidth, maxInputLength, maxSeqLength);
}

void initOutputIds(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    ITensor const& inputOffsets, TokenIdType const padId, TokenIdType const endId, SizeType32 const maxInputLength,
    bool const inputPacked, CudaStream const& stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    kernels::invokeFill(outputIds, endId, stream);

    if (inputPacked)
    {
        kernels::invokeCopyPackedInputToOutput(outputIds, inputIds, inputOffsets, maxInputLength, padId, stream);
    }
    else
    {
        kernels::invokeCopyInputToOutput(outputIds, inputIds, inputLengths, padId, stream);
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

namespace
{
template <typename T>
__global__ void scatterTensor(T* output, T const* input, std::uint32_t const batchSize,
    std::uint32_t const inputRowSize, std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = batchIdx * inputRowSize + columnIdx;
            auto const value = input[inputIdx];
            std::size_t constexpr beamIdx{0};
            auto const outputIdx = (batchIdx * beamWidth + beamIdx) * outputRowSize + columnIdx;
            output[outputIdx] = value;
        }
    }
}

template <typename T>
__global__ void splitTransposed(T* output, T const* input, std::uint32_t const batchSize,
    std::uint32_t const inputRowSize, std::uint32_t const split)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const tidz = static_cast<std::size_t>(blockIdx.z) * blockDim.z + threadIdx.z;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;
    auto const stridez = static_cast<std::size_t>(blockDim.z) * gridDim.z;

    auto const splitRowSize = static_cast<std::size_t>(inputRowSize / split);
    for (auto pIdx = tidz; pIdx < split; pIdx += stridez)
    {
        for (auto bid = tidx; bid < batchSize; bid += stridex)
        {
            for (auto colIdx = tidy; colIdx < splitRowSize; colIdx += stridey)
            {
                auto outputIdx
                    = common::flat_index3(pIdx, bid, colIdx, static_cast<std::size_t>(batchSize), splitRowSize);
                auto inputIdx
                    = common::flat_index2(bid, colIdx + pIdx * splitRowSize, static_cast<std::size_t>(inputRowSize));
                output[outputIdx] = input[inputIdx];
            }
        }
    }
}

template <typename T>
__global__ void tileTensor(T* output, T const* input, std::uint32_t const batchSize, std::size_t const inputRowSize,
    std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = batchIdx * inputRowSize + columnIdx;
            auto const value = input[inputIdx];
            for (std::size_t beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = (batchIdx * beamWidth + beamIdx) * outputRowSize + columnIdx;
                output[outputIdx] = value;
            }
        }
    }
}

template <typename T>
__global__ void tileTensorInPlace(
    T* inputOutput, std::uint32_t const batchSize, std::size_t const inputOutputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputOutputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = (batchIdx * beamWidth + 0) * inputOutputRowSize + columnIdx;
            auto const value = inputOutput[inputIdx];
            for (std::size_t beamIdx = 1; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = (batchIdx * beamWidth + beamIdx) * inputOutputRowSize + columnIdx;
                inputOutput[outputIdx] = value;
            }
        }
    }
}

} // namespace

template <typename T>
void invokeScatterTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    scatterTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input),
        nbInputRows, inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

void scatterTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeScatterTensor<SizeType32>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeScatterTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeScatterTensor<half>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeScatterTensor<int8_t>(output, input, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeScatterTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

template <typename T>
void invokeSplitTransposed(ITensor& output, ITensor const& input, SizeType32 split, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);
    auto const inputNbElems = input.getSize();
    auto const outputNbElems = output.getSize();

    TLLM_CHECK_WITH_INFO(
        nbOutputRows == split, common::fmtstr("nbOutputRows (%d) must be split (%d)", nbOutputRows, split));
    TLLM_CHECK_WITH_INFO(
        inputNbElems == outputNbElems, common::fmtstr("input and output must have the same number of elements"));

    dim3 const blockSize{256, 1, 1};
    std::size_t const gridx{tc::ceilDiv(nbInputRows, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{
        static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(inputRowSize), 1};
    splitTransposed<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<T>(output), bufferCast<T const>(input), nbInputRows, inputRowSize, static_cast<uint32_t>(split));
}

void splitTransposed(ITensor& output, ITensor const& input, SizeType32 split, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeSplitTransposed<SizeType32>(output, input, split, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeSplitTransposed<float>(output, input, split, stream); break;
    case nvinfer1::DataType::kHALF: invokeSplitTransposed<half>(output, input, split, stream); break;
    case nvinfer1::DataType::kINT8: invokeSplitTransposed<int8_t>(output, input, split, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeSplitTransposed<__hip_fp8_e4m3_fnuz>(output, input, split, stream); break;
#endif // ENABLE_FP8
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: invokeSplitTransposed<__hip_bfloat16>(output, input, split, stream); break;
#endif // ENABLE_BF16
    default: TLLM_THROW("data type not supported");
    }
}

template <typename T>
void invokeTileTensor(ITensor& output, ITensor const& input, SizeType32 const beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    tileTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input), nbInputRows,
        inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

void tileTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeTileTensor<SizeType32>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeTileTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeTileTensor<half>(output, input, beamWidth, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: invokeTileTensor<__hip_bfloat16>(output, input, beamWidth, stream); break;
#endif // ENABLE_BF16
    case nvinfer1::DataType::kINT8: invokeTileTensor<int8_t>(output, input, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeTileTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

template <typename T>
void invokeTileTensorInPlace(ITensor& inputOutput, SizeType32 const beamWidth, CudaStream const& stream)
{
    auto const& inputOutputShape = inputOutput.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(inputOutputShape.d[0]);
    auto const nbInputRows = nbOutputRows / static_cast<std::uint32_t>(beamWidth);
    auto const inputOutputRowSize = inputOutput.getSize() / static_cast<std::size_t>(nbOutputRows);

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputOutputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    tileTensorInPlace<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<T>(inputOutput), nbInputRows, inputOutputRowSize, static_cast<std::uint32_t>(beamWidth));
}

void tileTensorInplace(ITensor& tensor, SizeType32 beamWidth, CudaStream const& stream)
{
    switch (tensor.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeTileTensorInPlace<SizeType32>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeTileTensorInPlace<float>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeTileTensorInPlace<half>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeTileTensorInPlace<int8_t>(tensor, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeTileTensorInPlace<__hip_fp8_e4m3_fnuz>(tensor, beamWidth, stream); break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

// In the following kernel, we launch a grid with batchSize blocks of threads. Each thread block
// copies the logits from the "logits" tensor to the "lastTokenLogits" tensor for the last token
// of each sequence.
//
// TODO: Enable vector copies for higher BW utilization.

template <typename T>
__global__ void gatherLastTokenLogitsKernel(T* lastTokenLogits, T const* logits, int const* lastTokenIds,
    int maxInputLength, int beamWidth, int vocabSizePadded)
{
    // This sequence.
    int seqIdx = blockIdx.x;
    // Find the index of the last token in that sequence.
    // Since lastTokenIds is the accumulated length instead of real ids, so we need to minus 1.
    // For length [11, 23], we hope to get the results of id 10 and 22, in fact.
    int lastTokenIdx = lastTokenIds[seqIdx] - 1;

    // The output pointer.
    T* lastTokenLogitsPtr = &lastTokenLogits[seqIdx * beamWidth * vocabSizePadded];
    // The input pointer.
    T const* logitsPtr = &logits[lastTokenIdx * vocabSizePadded];

    // The threads in the block collaborate to copy the logits.
    for (int idx = threadIdx.x; idx < vocabSizePadded; idx += blockDim.x)
    {
        T value = logitsPtr[idx];
        for (int beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
        {
            lastTokenLogitsPtr[beamIdx * vocabSizePadded + idx] = value;
        }
    }
}

template <typename T>
void invokeGatherLastTokenLogits(
    ITensor& output, ITensor const& input, ITensor const& lastTokenIds, CudaStream const& stream)
{
    auto const& outputShape = output.getShape();
    auto const batchSize = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const beamWidth = static_cast<std::uint32_t>(outputShape.d[1]);
    auto const vocabSizePadded = static_cast<std::uint32_t>(outputShape.d[2]);

    auto const& inputShape = input.getShape();
    auto const maxInputLength = static_cast<std::uint32_t>(inputShape.d[1]);

    TLLM_CHECK_WITH_INFO(inputShape.d[0] == batchSize, "Invalid input shape: dim[0]");
    TLLM_CHECK_WITH_INFO(inputShape.d[2] == vocabSizePadded, "Invalid input shape: dim[2]");

    dim3 const blockSize{256, 1};
    dim3 const gridSize{static_cast<std::uint32_t>(batchSize), 1};
    gatherLastTokenLogitsKernel<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T>(input),
        bufferCast<int32_t>(lastTokenIds), static_cast<std::uint32_t>(maxInputLength),
        static_cast<std::uint32_t>(beamWidth), vocabSizePadded);
}

void gatherLastTokenLogits(ITensor& output, ITensor const& input, ITensor const& lastTokenIds, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kFLOAT: invokeGatherLastTokenLogits<float>(output, input, lastTokenIds, stream); break;
    case nvinfer1::DataType::kHALF: invokeGatherLastTokenLogits<half>(output, input, lastTokenIds, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        invokeGatherLastTokenLogits<__hip_bfloat16>(output, input, lastTokenIds, stream);
        break;
#endif // ENABLE_BF16
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8:
        invokeGatherLastTokenLogits<__hip_fp8_e4m3_fnuz>(output, input, lastTokenIds, stream);
        break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

namespace
{
// In the following kernel, we launch a grid with (microBatchSize * beamWidth, outputLen) blocks of threads. Each thread
// block copies a `vocabSizePadded` length logits tensor from the "inputLogits (microBatchSize, beamWidth,
// vocabSizePadded)" to the "outputGenerationLogits (batchSize, beamWidth, outputLen, vocabSizePadded)"
template <typename T>
__global__ void mergeLogitsFragmentsKernel(T* output, T** fragmentsVector, int const outputLen, int firstBatchSlotIdx,
    int microBatchSize, int beamWidth, int vocabSizePadded, int stepOffset)
{
    // output: shape: [batchSize, beamWidth, outputLen, vocabSize]
    // inputVecor.at(i): shape: [microBatchSize, beamWidth, vocabSize]

    // Current step
    int curStep = blockIdx.y;

    // The relatively batch slot index that this thread block in microBatchSize.
    int relativeBatchSlotIdx = blockIdx.x / beamWidth;

    // The Absolute batch slot index in batchSize.
    int absoluteBatchSlotIdx = firstBatchSlotIdx + relativeBatchSlotIdx;

    // The beam index that this thread block process
    int mbeamIdx = blockIdx.x % beamWidth;

    // The output pointer
    unsigned int const outputOffset
        = (absoluteBatchSlotIdx * beamWidth * outputLen + mbeamIdx * outputLen + curStep + stepOffset)
        * vocabSizePadded;

    T* outputPtr = &output[outputOffset];

    unsigned int const inputOffset = (relativeBatchSlotIdx * beamWidth + mbeamIdx) * vocabSizePadded;
    // The input pointer.
    T const* inputPtr = &fragmentsVector[curStep][inputOffset];

    // The threads in the block collaborate to copy the logits.
    for (int idx = threadIdx.x; idx < vocabSizePadded; idx += blockDim.x)
    {
        outputPtr[idx] = inputPtr[idx];
    }
}

template <typename T>
void invokeMergeLogitsFragments(BufferManager const& bufferManager, ITensor& output,
    std::vector<TensorPtr> const& fragmentsVector, ITensor& cachePointerDevice, ITensor& cachePointerHost,
    SizeType32 firstBatchSlotIdx, SizeType32 const microBatchSize, SizeType32 const beamWidth, CudaStream const& stream,
    int stepOffset)
{
    size_t fragmentsVectorSize = fragmentsVector.size();

    auto cachePointerHostPtr = bufferCast<T*>(cachePointerHost);

    for (int i = 0; i < fragmentsVectorSize; i++)
    {
        cachePointerHostPtr[i] = bufferCast<T>(*fragmentsVector.at(i));
    }
    bufferManager.copy(cachePointerHost, cachePointerDevice);

    dim3 blockSize(256);
    dim3 gridSize{(unsigned int) (microBatchSize * beamWidth), (unsigned int) (fragmentsVectorSize)};

    auto const& outputShape = output.getShape();
    auto const vocabSizePadded = static_cast<SizeType32>(outputShape.d[outputShape.nbDims - 1]);
    auto const outputLen = static_cast<SizeType32>(outputShape.d[outputShape.nbDims - 2]);

    TLLM_CHECK_WITH_INFO(outputLen >= fragmentsVectorSize, "Fragments size does not match outputLen size");

    mergeLogitsFragmentsKernel<T><<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output),
        bufferCast<T*>(cachePointerDevice), outputLen, firstBatchSlotIdx, microBatchSize, beamWidth, vocabSizePadded,
        stepOffset);
}
} // namespace

void mergeLogitsFragments(BufferManager const& bufferManager, ITensor& output,
    std::vector<TensorPtr> const& fragmentsVector, ITensor& cachePointerDevice, ITensor& cachePointerHost,
    SizeType32 firstBatchSlotIdx, SizeType32 const microBatchSize, SizeType32 const beamWidth, CudaStream const& stream,
    int stepOffset)
{
    switch (output.getDataType())
    {
    case nvinfer1::DataType::kFLOAT:
        invokeMergeLogitsFragments<float>(bufferManager, output, fragmentsVector, cachePointerDevice, cachePointerHost,
            firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
    case nvinfer1::DataType::kHALF:
        invokeMergeLogitsFragments<half>(bufferManager, output, fragmentsVector, cachePointerDevice, cachePointerHost,
            firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        invokeMergeLogitsFragments<__hip_bfloat16>(bufferManager, output, fragmentsVector, cachePointerDevice,
            cachePointerHost, firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#endif // ENABLE_BF16
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8:
        invokeMergeLogitsFragments<__hip_fp8_e4m3_fnuz>(bufferManager, output, fragmentsVector, cachePointerDevice,
            cachePointerHost, firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

void invokeUpdateKVBlockArrayDraftTokenLocation(ITensor const& seqAcceptedDraftTokenOffsets,
    ITensor const& packedAcceptedDraftTokensIndices, ITensor const& pastKeyValueLengths, void* const* pointerArray,
    ::tensorrt_llm::kernels::KVCacheIndex const* offsetArray, SizeType32 layerCount, SizeType32 seqCount,
    SizeType32 numKVHeads, SizeType32 sizeInBytesPerKVHead, SizeType32 rewindDraftTokenCommonCount,
    SizeType32 const* rewindDraftTokenSeparateAdjustments, ITensor const& seqSlotRemapping, ITensor const& batchSlots,
    SizeType32 maxKVCacheLen, SizeType32 maxBlocksPerSeq, SizeType32 tokensPerBlock, bool canUseOneMoreBlock,
    hipStream_t stream)
{
    tensorrt_llm::kernels::speculative_decoding::updateKVBlockArrayDraftTokenLocation(
        bufferCast<SizeType32>(seqAcceptedDraftTokenOffsets), bufferCast<SizeType32>(packedAcceptedDraftTokensIndices),
        bufferCast<SizeType32>(pastKeyValueLengths), pointerArray, offsetArray, layerCount, seqCount, numKVHeads,
        sizeInBytesPerKVHead, rewindDraftTokenCommonCount, rewindDraftTokenSeparateAdjustments,
        bufferCast<SizeType32>(seqSlotRemapping), bufferCast<SizeType32>(batchSlots), maxKVCacheLen, maxBlocksPerSeq,
        tokensPerBlock, canUseOneMoreBlock, stream);
}

} // namespace tensorrt_llm::runtime::kernels
