#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION &
 * AFFILIATES. All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/kernels/kvCacheIndex.h"
#include "tensorrt_llm/kernels/speculativeDecoding/kvCacheUpdateKernels.h"
#include "tensorrt_llm/runtime/runtimeKernels.h"

#include <NvInferRuntimeBase.h>
#include <hip/hip_runtime.h>

using namespace tensorrt_llm::runtime;
namespace tc = tensorrt_llm::common;

namespace tensorrt_llm::runtime::kernels
{

namespace
{

template <typename T>
__global__ void fill(T* data, std::size_t size, T const value)
{
    auto const tidx = (static_cast<std::size_t>(blockIdx.x) * blockDim.x) + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] = value;
    }
}

//! @param data    expected shape [indicesRange, size]
//! @param indices expected shape [gridDim.y]
//! @param size
//! @param values  expected shape [gridDim.y]
template <typename T>
__global__ void fillBatch(T* data, std::int32_t const* indices, std::size_t size, T const* values)
{
    auto const batchIdx = indices[blockIdx.y];
    T const value = values[blockIdx.y];
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const startIdx = batchIdx * size;
    auto const endIdx = startIdx + size;

    for (auto idx = startIdx + tidx; idx < endIdx; idx += stride)
    {
        data[idx] = value;
    }
}

template <typename T>
void invokeFillBatch(IBuffer& buffer, IBuffer const& slotIndices, std::size_t slotStride, IBuffer const& values,
    CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const* const indices = bufferCast<std::int32_t>(slotIndices);
    auto fillValues = bufferCast<T>(values);
    auto numSlots = slotIndices.getSize();
    auto const size = slotStride;
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};

    fillBatch<<<gridSize, blockSize, 0, stream.get()>>>(data, indices, size, fillValues);
}

//! @param data    expected shape [gridDim.y, size]
//! @param indices expected shape [gridDim.y]
//! @param size
//! @param values  expected shape [indicesRange, size]
template <typename T>
__global__ void gatherBatch(T* data, T const* values, std::int32_t const* indices, std::size_t size)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        auto const batchIdx = blockIdx.y;
        auto const slotIdx = indices[blockIdx.y];
        data[batchIdx + idx] = values[slotIdx + idx];
    }
}

template <typename T>
void invokeGatherBatch(IBuffer& buffer, IBuffer const& values, IBuffer const& slotIndices, std::size_t slotStride,
    CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const* const indices = bufferCast<std::int32_t>(slotIndices);
    auto sparseValues = bufferCast<T>(values);
    auto numSlots = slotIndices.getSize();
    auto const size = slotStride;
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};

    gatherBatch<<<gridSize, blockSize, 0, stream.get()>>>(data, sparseValues, indices, size);
}

template <typename VecT>
__global__ void copyBatch(uint8_t const* srcData, uint8_t* dstData, SizeType64 const* srcOffsets,
    SizeType64 const* dstOffsets, SizeType64 const* sizes, SizeType64 const dataTypeSize)
{
    constexpr auto VEC_ELTS = static_cast<int32_t>(sizeof(VecT));
    SizeType64 const srcStartIdx = srcOffsets[blockIdx.y] * dataTypeSize;
    SizeType64 const dstStartIdx = dstOffsets[blockIdx.y] * dataTypeSize;
    SizeType64 const size = sizes[blockIdx.y] * dataTypeSize;
    SizeType64 const tidx = (static_cast<SizeType64>(blockIdx.x) * blockDim.x + threadIdx.x) * VEC_ELTS;
    SizeType64 const stride = static_cast<SizeType64>(blockDim.x) * gridDim.x * VEC_ELTS;
    SizeType64 const srcEndIdx = srcStartIdx + size;

    SizeType64 srcIdx = srcStartIdx + tidx;
    SizeType64 dstIdx = dstStartIdx + tidx;

    for (; srcIdx < srcEndIdx; srcIdx += stride, dstIdx += stride)
    {
        *reinterpret_cast<VecT*>(&dstData[dstIdx]) = *reinterpret_cast<VecT const*>(&srcData[srcIdx]);
    }
}

template <typename T>
__global__ void scatterTensor(T* output, T const* input, std::uint32_t const batchSize,
    std::uint32_t const inputRowSize, std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = (static_cast<std::size_t>(blockIdx.x) * blockDim.x) + threadIdx.x;
    auto const tidy = (static_cast<std::size_t>(blockIdx.y) * blockDim.y) + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = (batchIdx * inputRowSize) + columnIdx;
            auto const value = input[inputIdx];
            std::size_t constexpr beamIdx{0};
            auto const outputIdx = ((batchIdx * beamWidth + beamIdx) * outputRowSize) + columnIdx;
            output[outputIdx] = value;
        }
    }
}

template <typename T>
__global__ void tileTensor(T* output, T const* input, std::uint32_t const batchSize, std::size_t const inputRowSize,
    std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = (static_cast<std::size_t>(blockIdx.x) * blockDim.x) + threadIdx.x;
    auto const tidy = (static_cast<std::size_t>(blockIdx.y) * blockDim.y) + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = (batchIdx * inputRowSize) + columnIdx;
            auto const value = input[inputIdx];
            for (std::size_t beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = ((batchIdx * beamWidth + beamIdx) * outputRowSize) + columnIdx;
                output[outputIdx] = value;
            }
        }
    }
}

template <typename T>
void invokeScatterTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    scatterTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input),
        nbInputRows, inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

template <typename T>
void invokeTileTensor(ITensor& output, ITensor const& input, SizeType32 const beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    tileTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input), nbInputRows,
        inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

// In the following kernel, we launch a grid with (microBatchSize * beamWidth, outputLen) blocks of threads. Each thread
// block copies a `vocabSizePadded` length logits tensor from the "inputLogits (microBatchSize, beamWidth,
// vocabSizePadded)" to the "outputGenerationLogits (batchSize, beamWidth, outputLen, vocabSizePadded)"
template <typename T>
__global__ void mergeLogitsFragmentsKernel(T* output, T** fragmentsVector, int const outputLen, int firstBatchSlotIdx,
    int beamWidth, int vocabSizePadded, int stepOffset)
{
    // output: shape: [batchSize, beamWidth, outputLen, vocabSize]
    // inputVecor.at(i): shape: [microBatchSize, beamWidth, vocabSize]

    // Current step
    int const curStep = blockIdx.y;

    // The relatively batch slot index that this thread block in microBatchSize.
    int const relativeBatchSlotIdx = blockIdx.x / beamWidth;

    // The Absolute batch slot index in batchSize.
    int const absoluteBatchSlotIdx = firstBatchSlotIdx + relativeBatchSlotIdx;

    // The beam index that this thread block process
    int const mbeamIdx = blockIdx.x % beamWidth;

    // The output pointer
    unsigned int const outputOffset
        = (absoluteBatchSlotIdx * beamWidth * outputLen + mbeamIdx * outputLen + curStep + stepOffset)
        * vocabSizePadded;

    T* outputPtr = &output[outputOffset];

    unsigned int const inputOffset = (relativeBatchSlotIdx * beamWidth + mbeamIdx) * vocabSizePadded;
    // The input pointer.
    T const* inputPtr = &fragmentsVector[curStep][inputOffset];

    // The threads in the block collaborate to copy the logits.
    for (int idx = threadIdx.x; idx < vocabSizePadded; idx += blockDim.x)
    {
        outputPtr[idx] = inputPtr[idx];
    }
}

template <typename T>
void invokeMergeLogitsFragments(BufferManager const& bufferManager, ITensor& output,
    std::vector<TensorPtr> const& fragmentsVector, ITensor& cachePointerDevice, ITensor& cachePointerHost,
    SizeType32 firstBatchSlotIdx, SizeType32 microBatchSize, SizeType32 beamWidth, CudaStream const& stream,
    int stepOffset)
{
    size_t const fragmentsVectorSize = fragmentsVector.size();

    auto cachePointerHostPtr = bufferCast<T*>(cachePointerHost);

    for (int i = 0; i < fragmentsVectorSize; i++)
    {
        cachePointerHostPtr[i] = bufferCast<T>(*fragmentsVector.at(i));
    }
    bufferManager.copy(cachePointerHost, cachePointerDevice);

    dim3 const blockSize(256);
    dim3 const gridSize{(unsigned int) (microBatchSize * beamWidth), (unsigned int) (fragmentsVectorSize)};

    auto const& outputShape = output.getShape();
    auto const vocabSizePadded = static_cast<SizeType32>(outputShape.d[outputShape.nbDims - 1]);
    auto const outputLen = static_cast<SizeType32>(outputShape.d[outputShape.nbDims - 2]);

    TLLM_CHECK_WITH_INFO(outputLen >= fragmentsVectorSize, "Fragments size does not match outputLen size");

    mergeLogitsFragmentsKernel<T><<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output),
        bufferCast<T*>(cachePointerDevice), outputLen, firstBatchSlotIdx, beamWidth, vocabSizePadded, stepOffset);
}

} // namespace

template <typename T>
void invokeFill(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    fill<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

// template instantiation
template void invokeFill(IBuffer&, SizeType64, CudaStream const&);
template void invokeFill(IBuffer&, std::int32_t, CudaStream const&);
template void invokeFill(IBuffer&, std::int8_t, CudaStream const&);
template void invokeFill(IBuffer&, std::uint8_t, CudaStream const&);
template void invokeFill(IBuffer&, bool, CudaStream const&);
template void invokeFill(IBuffer&, half, CudaStream const&);
template void invokeFill(IBuffer&, float, CudaStream const&);
#ifdef ENABLE_BF16
template void invokeFill(IBuffer&, __hip_bfloat16, CudaStream const&);
#endif // ENABLE_BF16

void invokeFillBatch(IBuffer& buffer, IBuffer const& slotIndices, std::size_t slotStride, IBuffer const& values,
    CudaStream const& stream)
{
    switch (buffer.getDataType())
    {
    case nvinfer1::DataType::kINT32:
        invokeFillBatch<std::int32_t>(buffer, slotIndices, slotStride, values, stream);
        break;
    case nvinfer1::DataType::kINT8:
        invokeFillBatch<std::int8_t>(buffer, slotIndices, slotStride, values, stream);
        break;
    case nvinfer1::DataType::kFLOAT: invokeFillBatch<float>(buffer, slotIndices, slotStride, values, stream); break;
    default: TLLM_THROW("data type not supported");
    }
}

void invokeGatherBatch(IBuffer& buffer, IBuffer const& values, IBuffer const& slotIndices, std::size_t slotStride,
    CudaStream const& stream)
{
    switch (buffer.getDataType())
    {
    case nvinfer1::DataType::kINT32:
        invokeGatherBatch<std::int32_t>(buffer, values, slotIndices, slotStride, stream);
        break;
    case nvinfer1::DataType::kINT8:
        invokeGatherBatch<std::int8_t>(buffer, values, slotIndices, slotStride, stream);
        break;
    case nvinfer1::DataType::kFLOAT: invokeGatherBatch<float>(buffer, values, slotIndices, slotStride, stream); break;
    default: TLLM_THROW("data type not supported");
    }
}

void invokeCopyBatch(IBuffer const& srcBuffer, IBuffer& dstBuffer, IBuffer const& srcOffsets, IBuffer const& dstOffsets,
    IBuffer const& sizes, std::size_t maxStride, CudaStream const& stream)
{
    auto const* srcDataPtr = reinterpret_cast<uint8_t const*>(srcBuffer.data());
    auto* dstDataPtr = reinterpret_cast<uint8_t*>(dstBuffer.data());
    auto const* srcOffsetsPtr = bufferCast<SizeType64>(srcOffsets);
    auto const* dstOffsetsPtr = bufferCast<SizeType64>(dstOffsets);
    auto const* sizesPtr = bufferCast<SizeType64>(sizes);
    auto numSlots = srcOffsets.getSize();
    auto const size = maxStride;
    auto const dataTypeSize = BufferDataType(srcBuffer.getDataType()).getSize();
    auto const copyRowSizeInBytes = size * dataTypeSize;

    auto copyBatchInvocation = copyBatch<uint8_t>;
    auto vectorSize = 1;
    if (dataTypeSize % 16 == 0)
    {
        vectorSize = 16;
        copyBatchInvocation = copyBatch<uint4>;
    }
    else if (dataTypeSize % 8 == 0)
    {
        vectorSize = 8;
        copyBatchInvocation = copyBatch<uint2>;
    }
    else if (dataTypeSize % 4 == 0)
    {
        vectorSize = 4;
        copyBatchInvocation = copyBatch<uint32_t>;
    }
    else if (dataTypeSize % 2 == 0)
    {
        vectorSize = 2;
        copyBatchInvocation = copyBatch<uint16_t>;
    }

    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(copyRowSizeInBytes / vectorSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};
    copyBatchInvocation<<<gridSize, blockSize, 0, stream.get()>>>(
        srcDataPtr, dstDataPtr, srcOffsetsPtr, dstOffsetsPtr, sizesPtr, static_cast<SizeType64>(dataTypeSize));
}

void scatterTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeScatterTensor<SizeType32>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeScatterTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeScatterTensor<half>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeScatterTensor<int8_t>(output, input, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeScatterTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

void tileTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeTileTensor<SizeType32>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeTileTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeTileTensor<half>(output, input, beamWidth, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: invokeTileTensor<__hip_bfloat16>(output, input, beamWidth, stream); break;
#endif // ENABLE_BF16
    case nvinfer1::DataType::kINT8: invokeTileTensor<int8_t>(output, input, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeTileTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

void mergeLogitsFragments(BufferManager const& bufferManager, ITensor& output,
    std::vector<TensorPtr> const& fragmentsVector, ITensor& cachePointerDevice, ITensor& cachePointerHost,
    SizeType32 firstBatchSlotIdx, SizeType32 const microBatchSize, SizeType32 const beamWidth, CudaStream const& stream,
    int stepOffset)
{
    switch (output.getDataType())
    {
    case nvinfer1::DataType::kFLOAT:
        invokeMergeLogitsFragments<float>(bufferManager, output, fragmentsVector, cachePointerDevice, cachePointerHost,
            firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
    case nvinfer1::DataType::kHALF:
        invokeMergeLogitsFragments<half>(bufferManager, output, fragmentsVector, cachePointerDevice, cachePointerHost,
            firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        invokeMergeLogitsFragments<__hip_bfloat16>(bufferManager, output, fragmentsVector, cachePointerDevice,
            cachePointerHost, firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#endif // ENABLE_BF16
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8:
        invokeMergeLogitsFragments<__hip_fp8_e4m3_fnuz>(bufferManager, output, fragmentsVector, cachePointerDevice,
            cachePointerHost, firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

void invokeUpdateKVBlockArrayDraftTokenLocation(ITensor const& seqAcceptedDraftTokenOffsets,
    ITensor const& packedAcceptedDraftTokensIndices, ITensor const& pastKeyValueLengths, void* const* pointerArray,
    ::tensorrt_llm::kernels::KVCacheIndex const* offsetArray, SizeType32 layerCount, SizeType32 seqCount,
    SizeType32 numKVHeads, SizeType32 sizeInBytesPerKVHead, SizeType32 rewindDraftTokenCommonCount,
    SizeType32 const* rewindDraftTokenSeparateAdjustments, ITensor const& seqSlotRemapping, ITensor const& batchSlots,
    SizeType32 maxKVCacheLen, SizeType32 maxBlocksPerSeq, SizeType32 tokensPerBlock, bool canUseOneMoreBlock,
    hipStream_t stream)
{
    tensorrt_llm::kernels::speculative_decoding::updateKVBlockArrayDraftTokenLocation(
        bufferCast<SizeType32>(seqAcceptedDraftTokenOffsets), bufferCast<SizeType32>(packedAcceptedDraftTokensIndices),
        bufferCast<SizeType32>(pastKeyValueLengths), pointerArray, offsetArray, layerCount, seqCount, numKVHeads,
        sizeInBytesPerKVHead, rewindDraftTokenCommonCount, rewindDraftTokenSeparateAdjustments,
        bufferCast<SizeType32>(seqSlotRemapping), bufferCast<SizeType32>(batchSlots), maxKVCacheLen, maxBlocksPerSeq,
        tokensPerBlock, canUseOneMoreBlock, stream);
}

} // namespace tensorrt_llm::runtime::kernels
