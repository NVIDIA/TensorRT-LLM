#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION &
 * AFFILIATES. All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/speculativeDecoding/kvCacheUpdateKernels.h"
#include "tensorrt_llm/runtime/runtimeKernels.h"

#include <NvInferRuntimeBase.h>
#include <hipcub/hipcub.hpp>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

using namespace tensorrt_llm::runtime;
namespace tc = tensorrt_llm::common;

namespace tensorrt_llm::runtime::kernels
{

namespace
{

template <typename T>
__global__ void fill(T* data, std::size_t size, T const value)
{
    auto const tidx = (static_cast<std::size_t>(blockIdx.x) * blockDim.x) + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] = value;
    }
}

//! @param data    expected shape [indicesRange, size]
//! @param indices expected shape [gridDim.y]
//! @param size
//! @param values  expected shape [gridDim.y]
template <typename T>
__global__ void fillBatch(T* data, std::int32_t const* indices, std::size_t size, T const* values)
{
    auto const batchIdx = indices[blockIdx.y];
    T const value = values[blockIdx.y];
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const startIdx = batchIdx * size;
    auto const endIdx = startIdx + size;

    for (auto idx = startIdx + tidx; idx < endIdx; idx += stride)
    {
        data[idx] = value;
    }
}

template <typename T>
void invokeFillBatch(IBuffer& buffer, IBuffer const& slotIndices, std::size_t slotStride, IBuffer const& values,
    CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const* const indices = bufferCast<std::int32_t>(slotIndices);
    auto fillValues = bufferCast<T>(values);
    auto numSlots = slotIndices.getSize();
    auto const size = slotStride;
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};

    fillBatch<<<gridSize, blockSize, 0, stream.get()>>>(data, indices, size, fillValues);
}

//! @param data    expected shape [gridDim.y, size]
//! @param indices expected shape [gridDim.y]
//! @param size
//! @param values  expected shape [indicesRange, size]
template <typename T>
__global__ void gatherBatch(T* data, T const* values, std::int32_t const* indices, std::size_t size)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        auto const batchIdx = blockIdx.y;
        auto const slotIdx = indices[blockIdx.y];
        data[batchIdx + idx] = values[slotIdx + idx];
    }
}

template <typename T>
void invokeGatherBatch(IBuffer& buffer, IBuffer const& values, IBuffer const& slotIndices, std::size_t slotStride,
    CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const* const indices = bufferCast<std::int32_t>(slotIndices);
    auto sparseValues = bufferCast<T>(values);
    auto numSlots = slotIndices.getSize();
    auto const size = slotStride;
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};

    gatherBatch<<<gridSize, blockSize, 0, stream.get()>>>(data, sparseValues, indices, size);
}

template <typename VecT>
__global__ void copyBatch(uint8_t const* srcData, uint8_t* dstData, SizeType64 const* srcOffsets,
    SizeType64 const* dstOffsets, SizeType64 const* sizes, SizeType64 const dataTypeSize)
{
    constexpr auto VEC_ELTS = static_cast<int32_t>(sizeof(VecT));
    SizeType64 const srcStartIdx = srcOffsets[blockIdx.y] * dataTypeSize;
    SizeType64 const dstStartIdx = dstOffsets[blockIdx.y] * dataTypeSize;
    SizeType64 const size = sizes[blockIdx.y] * dataTypeSize;
    SizeType64 const tidx = (static_cast<SizeType64>(blockIdx.x) * blockDim.x + threadIdx.x) * VEC_ELTS;
    SizeType64 const stride = static_cast<SizeType64>(blockDim.x) * gridDim.x * VEC_ELTS;
    SizeType64 const srcEndIdx = srcStartIdx + size;

    SizeType64 srcIdx = srcStartIdx + tidx;
    SizeType64 dstIdx = dstStartIdx + tidx;

    for (; srcIdx < srcEndIdx; srcIdx += stride, dstIdx += stride)
    {
        *reinterpret_cast<VecT*>(&dstData[dstIdx]) = *reinterpret_cast<VecT const*>(&srcData[srcIdx]);
    }
}

template <typename T>
__global__ void add(T* data, std::size_t size, T const value)
{
    auto const tidx = (static_cast<std::size_t>(blockIdx.x) * blockDim.x) + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] += value;
    }
}

template <typename T>
__global__ void reduceSum(T* output, T const* input, std::size_t size)
{
    T threadSum = 0;
    for (auto index = threadIdx.x; index < size; index += blockDim.x)
    {
        threadSum += input[index];
    }

    T blockSum = 0;
    if (blockDim.x <= 32)
    {
        blockSum = tc::warpReduceSum(threadSum);
    }
    else
    {
        blockSum = tc::blockReduceSum(threadSum);
    }
    __syncthreads();

    if (threadIdx.x == 0)
    {
        *output = blockSum;
    }
}

template <typename T>
void invokeReduce(IBuffer& output, IBuffer const& input, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");
    TLLM_CHECK_WITH_INFO(output.getSize() == 1, common::fmtstr("Output size (%ld) has to be 1", output.getSize()));

    auto outputPtr = bufferCast<T>(output);
    auto inputPtr = bufferCast<T>(input);
    auto const size = input.getSize();

    dim3 const blockSize{std::min(512U, static_cast<std::uint32_t>(size))};
    dim3 const gridSize{1};

    reduceSum<<<gridSize, blockSize, 0, stream.get()>>>(outputPtr, inputPtr, size);
}

__global__ void transposeWithOutputOffset(SizeType32* output, SizeType32 const* input, SizeType32 const nbInputRows,
    SizeType32 const inputRowSize, SizeType32 const outputRowSize, SizeType32 const outputOffset)
{
    SizeType32 const tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
    SizeType32 const tidy = (blockIdx.y * blockDim.y) + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < nbInputRows; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType32 tokenIdx = tidx; tokenIdx < inputRowSize; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = (batchIdx * inputRowSize) + tokenIdx;
            auto const outputIdx = (tokenIdx * outputRowSize) + outputOffset + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}

__global__ void buildAttentionMask(SizeType32* attentionMask, SizeType32 const size, SizeType32 const padId)
{
    SizeType32 const tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    for (SizeType32 i = tid; i < size; i += blockDim.x * gridDim.x)
    {
        auto const x = attentionMask[i];
        attentionMask[i] = (x != padId);
    }
}

__global__ void extendAttentionMask(
    SizeType32* newMask, SizeType32 const* oldMask, SizeType32 const batchSize, SizeType32 const seqLength)
{
    SizeType32 const tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
    SizeType32 const tidy = (blockIdx.y * blockDim.y) + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType32 tokenIdx = tidx; tokenIdx < seqLength + 1; tokenIdx += blockDim.x * gridDim.x)
        {
            SizeType32 const oldIndex = (batchIdx * seqLength) + tokenIdx;
            SizeType32 const newIndex = (batchIdx * (seqLength + 1)) + tokenIdx;
            newMask[newIndex] = (tokenIdx < seqLength) ? oldMask[oldIndex] : 1;
        }
    }
}

__global__ void copyInputToOutput(TokenIdType* outputIds, TokenIdType const* inputIds, SizeType32 const* inputLengths,
    TokenIdType const padId, SizeType32 const batchSize, SizeType32 const beamWidth, SizeType32 const maxInputLength,
    SizeType32 const maxSeqLength)
{
    SizeType32 const tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
    SizeType32 const tidy = (blockIdx.y * blockDim.y) + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType32 tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[(batchIdx * maxInputLength) + tokenIdx] : padId;
            for (SizeType32 beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(batchIdx, beamIdx, tokenIdx, beamWidth, maxSeqLength);
                outputIds[outputIdx] = value;
            }
        }
    }
}

// In the following kernel, we launch a grid with batchSize blocks of threads. Each thread block
// copies the logits from the "logits" tensor to the "lastTokenLogits" tensor for the last token
// of each sequence.
//
// TODO: Enable vector copies for higher BW utilization.

template <typename T>
__global__ void gatherLastTokenLogitsKernel(
    T* lastTokenLogits, T const* logits, int const* lastTokenIds, int beamWidth, int vocabSizePadded)
{
    // This sequence.
    int const seqIdx = blockIdx.x;
    // Find the index of the last token in that sequence.
    // Since lastTokenIds is the accumulated length instead of real ids, so we need to minus 1.
    // For length [11, 23], we hope to get the results of id 10 and 22, in fact.
    int const lastTokenIdx = lastTokenIds[seqIdx] - 1;

    // The output pointer.
    T* lastTokenLogitsPtr = &lastTokenLogits[seqIdx * beamWidth * vocabSizePadded];
    // The input pointer.
    T const* logitsPtr = &logits[lastTokenIdx * vocabSizePadded];

    // The threads in the block collaborate to copy the logits.
    for (int idx = threadIdx.x; idx < vocabSizePadded; idx += blockDim.x)
    {
        T value = logitsPtr[idx];
        for (int beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
        {
            lastTokenLogitsPtr[(beamIdx * vocabSizePadded) + idx] = value;
        }
    }
}

template <typename T>
void invokeGatherLastTokenLogits(
    ITensor& output, ITensor const& input, ITensor const& lastTokenIds, CudaStream const& stream)
{
    auto const& outputShape = output.getShape();
    auto const batchSize = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const beamWidth = static_cast<std::uint32_t>(outputShape.d[1]);
    auto const vocabSizePadded = static_cast<std::uint32_t>(outputShape.d[2]);

    auto const& inputShape = input.getShape();

    TLLM_CHECK_WITH_INFO(inputShape.d[0] == batchSize, "Invalid input shape: dim[0]");
    TLLM_CHECK_WITH_INFO(inputShape.d[2] == vocabSizePadded, "Invalid input shape: dim[2]");

    dim3 const blockSize{256, 1};
    dim3 const gridSize{static_cast<std::uint32_t>(batchSize), 1};
    gatherLastTokenLogitsKernel<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T>(input),
        bufferCast<int32_t>(lastTokenIds), static_cast<std::uint32_t>(beamWidth), vocabSizePadded);
}

__global__ void copyPackedInputToOutput(TokenIdType* outputIds, TokenIdType const* inputIds,
    SizeType32 const* inputOffsets, TokenIdType const padId, SizeType32 const batchSize, SizeType32 const beamWidth,
    SizeType32 const maxInputLength, SizeType32 const maxSeqLength)
{
    SizeType32 const tidx = (blockIdx.x * blockDim.x) + threadIdx.x;
    SizeType32 const tidy = (blockIdx.y * blockDim.y) + threadIdx.y;

    for (SizeType32 batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const tokenBegin = inputOffsets[batchIdx];
        auto const tokenEnd = inputOffsets[batchIdx + 1];
        auto const inputLength = tokenEnd - tokenBegin;

        for (SizeType32 tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[tokenBegin + tokenIdx] : padId;
            for (SizeType32 beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(batchIdx, beamIdx, tokenIdx, beamWidth, maxSeqLength);
                outputIds[outputIdx] = value;
            }
        }
    }
}

template <typename T>
__global__ void scatterTensor(T* output, T const* input, std::uint32_t const batchSize,
    std::uint32_t const inputRowSize, std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = (static_cast<std::size_t>(blockIdx.x) * blockDim.x) + threadIdx.x;
    auto const tidy = (static_cast<std::size_t>(blockIdx.y) * blockDim.y) + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = (batchIdx * inputRowSize) + columnIdx;
            auto const value = input[inputIdx];
            std::size_t constexpr beamIdx{0};
            auto const outputIdx = ((batchIdx * beamWidth + beamIdx) * outputRowSize) + columnIdx;
            output[outputIdx] = value;
        }
    }
}

template <typename T>
__global__ void tileTensor(T* output, T const* input, std::uint32_t const batchSize, std::size_t const inputRowSize,
    std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = (static_cast<std::size_t>(blockIdx.x) * blockDim.x) + threadIdx.x;
    auto const tidy = (static_cast<std::size_t>(blockIdx.y) * blockDim.y) + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = (batchIdx * inputRowSize) + columnIdx;
            auto const value = input[inputIdx];
            for (std::size_t beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = ((batchIdx * beamWidth + beamIdx) * outputRowSize) + columnIdx;
                output[outputIdx] = value;
            }
        }
    }
}

template <typename T>
void invokeScatterTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    scatterTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input),
        nbInputRows, inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

template <typename T>
void invokeTileTensor(ITensor& output, ITensor const& input, SizeType32 const beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    tileTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input), nbInputRows,
        inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

// In the following kernel, we launch a grid with (microBatchSize * beamWidth, outputLen) blocks of threads. Each thread
// block copies a `vocabSizePadded` length logits tensor from the "inputLogits (microBatchSize, beamWidth,
// vocabSizePadded)" to the "outputGenerationLogits (batchSize, beamWidth, outputLen, vocabSizePadded)"
template <typename T>
__global__ void mergeLogitsFragmentsKernel(T* output, T** fragmentsVector, int const outputLen, int firstBatchSlotIdx,
    int beamWidth, int vocabSizePadded, int stepOffset)
{
    // output: shape: [batchSize, beamWidth, outputLen, vocabSize]
    // inputVecor.at(i): shape: [microBatchSize, beamWidth, vocabSize]

    // Current step
    int const curStep = blockIdx.y;

    // The relatively batch slot index that this thread block in microBatchSize.
    int const relativeBatchSlotIdx = blockIdx.x / beamWidth;

    // The Absolute batch slot index in batchSize.
    int const absoluteBatchSlotIdx = firstBatchSlotIdx + relativeBatchSlotIdx;

    // The beam index that this thread block process
    int const mbeamIdx = blockIdx.x % beamWidth;

    // The output pointer
    unsigned int const outputOffset
        = (absoluteBatchSlotIdx * beamWidth * outputLen + mbeamIdx * outputLen + curStep + stepOffset)
        * vocabSizePadded;

    T* outputPtr = &output[outputOffset];

    unsigned int const inputOffset = (relativeBatchSlotIdx * beamWidth + mbeamIdx) * vocabSizePadded;
    // The input pointer.
    T const* inputPtr = &fragmentsVector[curStep][inputOffset];

    // The threads in the block collaborate to copy the logits.
    for (int idx = threadIdx.x; idx < vocabSizePadded; idx += blockDim.x)
    {
        outputPtr[idx] = inputPtr[idx];
    }
}

template <typename T>
void invokeMergeLogitsFragments(BufferManager const& bufferManager, ITensor& output,
    std::vector<TensorPtr> const& fragmentsVector, ITensor& cachePointerDevice, ITensor& cachePointerHost,
    SizeType32 firstBatchSlotIdx, SizeType32 microBatchSize, SizeType32 beamWidth, CudaStream const& stream,
    int stepOffset)
{
    size_t const fragmentsVectorSize = fragmentsVector.size();

    auto cachePointerHostPtr = bufferCast<T*>(cachePointerHost);

    for (int i = 0; i < fragmentsVectorSize; i++)
    {
        cachePointerHostPtr[i] = bufferCast<T>(*fragmentsVector.at(i));
    }
    bufferManager.copy(cachePointerHost, cachePointerDevice);

    dim3 const blockSize(256);
    dim3 const gridSize{(unsigned int) (microBatchSize * beamWidth), (unsigned int) (fragmentsVectorSize)};

    auto const& outputShape = output.getShape();
    auto const vocabSizePadded = static_cast<SizeType32>(outputShape.d[outputShape.nbDims - 1]);
    auto const outputLen = static_cast<SizeType32>(outputShape.d[outputShape.nbDims - 2]);

    TLLM_CHECK_WITH_INFO(outputLen >= fragmentsVectorSize, "Fragments size does not match outputLen size");

    mergeLogitsFragmentsKernel<T><<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output),
        bufferCast<T*>(cachePointerDevice), outputLen, firstBatchSlotIdx, beamWidth, vocabSizePadded, stepOffset);
}

void invokeCopyPackedInputToOutput(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputOffsets,
    SizeType32 const maxInputLength, TokenIdType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const& outputShape = outputIds.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 3, common::fmtstr("Output shape must have 3 dimensions, but has %d", outputShape.nbDims));

    auto const batchSize = static_cast<SizeType32>(inputOffsets.getSize()) - 1;
    SizeType32 const beamWidth = outputShape.d[1];
    SizeType32 const maxSeqLength = outputShape.d[2];

    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[0],
        common::fmtstr("Output ids batch size (" FMT_DIM ") does not match inputOffsets batch size (%d)",
            outputShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyPackedInputToOutput<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<TokenIdType>(outputIds),
        bufferCast<TokenIdType const>(inputIds), bufferCast<SizeType32 const>(inputOffsets), padId, batchSize,
        beamWidth, maxInputLength, maxSeqLength);
}

} // namespace

template <typename T>
void invokeFill(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    fill<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

// template instantiation
template void invokeFill(IBuffer&, SizeType64, CudaStream const&);
template void invokeFill(IBuffer&, std::int32_t, CudaStream const&);
template void invokeFill(IBuffer&, std::int8_t, CudaStream const&);
template void invokeFill(IBuffer&, std::uint8_t, CudaStream const&);
template void invokeFill(IBuffer&, bool, CudaStream const&);
template void invokeFill(IBuffer&, half, CudaStream const&);
template void invokeFill(IBuffer&, float, CudaStream const&);
#ifdef ENABLE_BF16
template void invokeFill(IBuffer&, __hip_bfloat16, CudaStream const&);
#endif // ENABLE_BF16

void invokeFillBatch(IBuffer& buffer, IBuffer const& slotIndices, std::size_t slotStride, IBuffer const& values,
    CudaStream const& stream)
{
    switch (buffer.getDataType())
    {
    case nvinfer1::DataType::kINT32:
        invokeFillBatch<std::int32_t>(buffer, slotIndices, slotStride, values, stream);
        break;
    case nvinfer1::DataType::kINT8:
        invokeFillBatch<std::int8_t>(buffer, slotIndices, slotStride, values, stream);
        break;
    case nvinfer1::DataType::kFLOAT: invokeFillBatch<float>(buffer, slotIndices, slotStride, values, stream); break;
    default: TLLM_THROW("data type not supported");
    }
}

void invokeGatherBatch(IBuffer& buffer, IBuffer const& values, IBuffer const& slotIndices, std::size_t slotStride,
    CudaStream const& stream)
{
    switch (buffer.getDataType())
    {
    case nvinfer1::DataType::kINT32:
        invokeGatherBatch<std::int32_t>(buffer, values, slotIndices, slotStride, stream);
        break;
    case nvinfer1::DataType::kINT8:
        invokeGatherBatch<std::int8_t>(buffer, values, slotIndices, slotStride, stream);
        break;
    case nvinfer1::DataType::kFLOAT: invokeGatherBatch<float>(buffer, values, slotIndices, slotStride, stream); break;
    default: TLLM_THROW("data type not supported");
    }
}

void invokeCopyBatch(IBuffer const& srcBuffer, IBuffer& dstBuffer, IBuffer const& srcOffsets, IBuffer const& dstOffsets,
    IBuffer const& sizes, std::size_t maxStride, CudaStream const& stream)
{
    auto const* srcDataPtr = reinterpret_cast<uint8_t const*>(srcBuffer.data());
    auto* dstDataPtr = reinterpret_cast<uint8_t*>(dstBuffer.data());
    auto const* srcOffsetsPtr = bufferCast<SizeType64>(srcOffsets);
    auto const* dstOffsetsPtr = bufferCast<SizeType64>(dstOffsets);
    auto const* sizesPtr = bufferCast<SizeType64>(sizes);
    auto numSlots = srcOffsets.getSize();
    auto const size = maxStride;
    auto const dataTypeSize = BufferDataType(srcBuffer.getDataType()).getSize();
    auto const copyRowSizeInBytes = size * dataTypeSize;

    auto copyBatchInvocation = copyBatch<uint8_t>;
    auto vectorSize = 1;
    if (dataTypeSize % 16 == 0)
    {
        vectorSize = 16;
        copyBatchInvocation = copyBatch<uint4>;
    }
    else if (dataTypeSize % 8 == 0)
    {
        vectorSize = 8;
        copyBatchInvocation = copyBatch<uint2>;
    }
    else if (dataTypeSize % 4 == 0)
    {
        vectorSize = 4;
        copyBatchInvocation = copyBatch<uint32_t>;
    }
    else if (dataTypeSize % 2 == 0)
    {
        vectorSize = 2;
        copyBatchInvocation = copyBatch<uint16_t>;
    }

    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(copyRowSizeInBytes / vectorSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), static_cast<std::uint32_t>(numSlots)};
    copyBatchInvocation<<<gridSize, blockSize, 0, stream.get()>>>(
        srcDataPtr, dstDataPtr, srcOffsetsPtr, dstOffsetsPtr, sizesPtr, static_cast<SizeType64>(dataTypeSize));
}

template <typename T>
void invokeAdd(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    add<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

template void invokeAdd(IBuffer&, std::int32_t, CudaStream const&);
template void invokeAdd(IBuffer&, std::int8_t, CudaStream const&);
template void invokeAdd(IBuffer&, float, CudaStream const&);

void reduce(IBuffer& output, IBuffer const& input, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeReduce<SizeType32>(output, input, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeReduce<float>(output, input, stream); break;
    case nvinfer1::DataType::kHALF: invokeReduce<half>(output, input, stream); break;
    case nvinfer1::DataType::kINT8: invokeReduce<int8_t>(output, input, stream); break;
    default: TLLM_THROW("data type not supported");
    }
}

void invokeTransposeWithOutputOffset(
    ITensor& output, ITensor const& input, SizeType32 const outputOffset, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));
    SizeType32 const nbInputRows = inputShape.d[0];
    SizeType32 const inputRowSize = inputShape.d[1];

    auto const& outputShape = output.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 2, common::fmtstr("Output shape must have 2 dimensions, but has %d", outputShape.nbDims));
    SizeType32 const nbOutputRows = outputShape.d[0];
    SizeType32 const outputRowSize = outputShape.d[1];

    TLLM_CHECK_WITH_INFO(inputRowSize == nbOutputRows,
        common::fmtstr("Input dim 1 (%d) and output dim 0 (%d) differ", inputRowSize, nbOutputRows));
    TLLM_CHECK_WITH_INFO(outputOffset + nbInputRows <= outputRowSize,
        common::fmtstr("Input (%d rows) does not fit into output (%d columns, offset %d)", nbInputRows, inputRowSize,
            outputOffset));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((inputRowSize + blockSize.x - 1) / blockSize.x, nbInputRows);

    transposeWithOutputOffset<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType32>(output),
        bufferCast<SizeType32 const>(input), nbInputRows, inputRowSize, outputRowSize, outputOffset);
}

void invokeInclusiveSum(IBuffer& output, IBuffer const& input, BufferManager const& manager, CudaStream const& stream)
{
    auto const size = input.getSize();
    auto const* inputData = bufferCast<SizeType32>(input);
    auto* outputData = bufferCast<SizeType32>(output);

    std::size_t tempStorageBytes{0};
    hipcub::DeviceScan::InclusiveSum(nullptr, tempStorageBytes, inputData, outputData, size, stream.get());
    auto tempStorage = manager.gpu(tempStorageBytes, nvinfer1::DataType::kUINT8);
    auto* tempStorageData = bufferCast<std::uint8_t>(*tempStorage);
    hipcub::DeviceScan::InclusiveSum(tempStorageData, tempStorageBytes, inputData, outputData, size, stream.get());
}

void invokeBuildAttentionMask(ITensor& attentionMask, SizeType32 const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        TRTDataType<SizeType32>::value == attentionMask.getDataType(), "attentionMask has wrong data type");

    auto const size = attentionMask.getSize();
    dim3 const blockSize(256);
    dim3 const gridSize((size + blockSize.x - 1) / blockSize.x);

    buildAttentionMask<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType32>(attentionMask), size, padId);
}

void invokeExtendAttentionMask(ITensor& newMask, ITensor const& oldMask, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType32>::value == newMask.getDataType(), "attentionMask has wrong data type");
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType32>::value == oldMask.getDataType(), "attentionMask has wrong data type");

    auto const& shape = oldMask.getShape();
    SizeType32 const batchSize = shape.d[0];
    SizeType32 const seqLength = shape.d[1];

    dim3 const blockSize(256, 1);
    dim3 const gridSize((seqLength + blockSize.x - 1) / blockSize.x, batchSize);

    extendAttentionMask<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<SizeType32>(newMask), bufferCast<SizeType32>(oldMask), batchSize, seqLength);
}

void invokeCopyInputToOutput(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    TokenIdType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const& inputShape = inputIds.getShape();
    auto const& outputShape = outputIds.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 3, common::fmtstr("Output shape must have 3 dimensions, but has %d", outputShape.nbDims));

    auto const batchSize = static_cast<SizeType32>(inputLengths.getSize());
    SizeType32 const maxInputLength = inputShape.d[inputShape.nbDims - 1];
    SizeType32 const beamWidth = outputShape.d[1];
    SizeType32 const maxSeqLength = outputShape.d[2];

    auto const inputBatchSize = inputIds.getSize() / maxInputLength;
    TLLM_CHECK_WITH_INFO(std::size_t(batchSize) == inputBatchSize,
        common::fmtstr("Input ids batch size (%ld) does not match inputLengths size (%ld)", inputBatchSize,
            std::size_t(batchSize)));
    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[0],
        common::fmtstr(
            "Output ids batch size (" FMT_DIM ") does not match inputLengths size (%d)", outputShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyInputToOutput<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<TokenIdType>(outputIds),
        bufferCast<TokenIdType const>(inputIds), bufferCast<SizeType32 const>(inputLengths), padId, batchSize,
        beamWidth, maxInputLength, maxSeqLength);
}

void initOutputIds(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    ITensor const& inputOffsets, TokenIdType const padId, TokenIdType const endId, SizeType32 const maxInputLength,
    bool const inputPacked, CudaStream const& stream)
{
    TLLM_LOG_TRACE("%s start", __PRETTY_FUNCTION__);
    kernels::invokeFill(outputIds, endId, stream);

    if (inputPacked)
    {
        invokeCopyPackedInputToOutput(outputIds, inputIds, inputOffsets, maxInputLength, padId, stream);
    }
    else
    {
        kernels::invokeCopyInputToOutput(outputIds, inputIds, inputLengths, padId, stream);
    }
    TLLM_LOG_TRACE("%s stop", __PRETTY_FUNCTION__);
}

void scatterTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeScatterTensor<SizeType32>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeScatterTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeScatterTensor<half>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeScatterTensor<int8_t>(output, input, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeScatterTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

void tileTensor(ITensor& output, ITensor const& input, SizeType32 beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeTileTensor<SizeType32>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeTileTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeTileTensor<half>(output, input, beamWidth, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16: invokeTileTensor<__hip_bfloat16>(output, input, beamWidth, stream); break;
#endif // ENABLE_BF16
    case nvinfer1::DataType::kINT8: invokeTileTensor<int8_t>(output, input, beamWidth, stream); break;
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8: invokeTileTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

void gatherLastTokenLogits(ITensor& output, ITensor const& input, ITensor const& lastTokenIds, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kFLOAT: invokeGatherLastTokenLogits<float>(output, input, lastTokenIds, stream); break;
    case nvinfer1::DataType::kHALF: invokeGatherLastTokenLogits<half>(output, input, lastTokenIds, stream); break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        invokeGatherLastTokenLogits<__hip_bfloat16>(output, input, lastTokenIds, stream);
        break;
#endif // ENABLE_BF16
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8:
        invokeGatherLastTokenLogits<__hip_fp8_e4m3_fnuz>(output, input, lastTokenIds, stream);
        break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

void mergeLogitsFragments(BufferManager const& bufferManager, ITensor& output,
    std::vector<TensorPtr> const& fragmentsVector, ITensor& cachePointerDevice, ITensor& cachePointerHost,
    SizeType32 firstBatchSlotIdx, SizeType32 const microBatchSize, SizeType32 const beamWidth, CudaStream const& stream,
    int stepOffset)
{
    switch (output.getDataType())
    {
    case nvinfer1::DataType::kFLOAT:
        invokeMergeLogitsFragments<float>(bufferManager, output, fragmentsVector, cachePointerDevice, cachePointerHost,
            firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
    case nvinfer1::DataType::kHALF:
        invokeMergeLogitsFragments<half>(bufferManager, output, fragmentsVector, cachePointerDevice, cachePointerHost,
            firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#ifdef ENABLE_BF16
    case nvinfer1::DataType::kBF16:
        invokeMergeLogitsFragments<__hip_bfloat16>(bufferManager, output, fragmentsVector, cachePointerDevice,
            cachePointerHost, firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#endif // ENABLE_BF16
#ifdef ENABLE_FP8
    case nvinfer1::DataType::kFP8:
        invokeMergeLogitsFragments<__hip_fp8_e4m3_fnuz>(bufferManager, output, fragmentsVector, cachePointerDevice,
            cachePointerHost, firstBatchSlotIdx, microBatchSize, beamWidth, stream, stepOffset);
        break;
#endif // ENABLE_FP8
    default: TLLM_THROW("data type not supported");
    }
}

void invokeUpdateKVBlockArrayDraftTokenLocation(ITensor const& seqAcceptedDraftTokenOffsets,
    ITensor const& packedAcceptedDraftTokensIndices, ITensor const& pastKeyValueLengths, void* const* pointerArray,
    ::tensorrt_llm::kernels::KVCacheIndex const* offsetArray, SizeType32 layerCount, SizeType32 seqCount,
    SizeType32 numKVHeads, SizeType32 sizeInBytesPerKVHead, SizeType32 rewindDraftTokenCommonCount,
    SizeType32 const* rewindDraftTokenSeparateAdjustments, ITensor const& seqSlotRemapping, ITensor const& batchSlots,
    SizeType32 maxKVCacheLen, SizeType32 maxBlocksPerSeq, SizeType32 tokensPerBlock, bool canUseOneMoreBlock,
    hipStream_t stream)
{
    tensorrt_llm::kernels::speculative_decoding::updateKVBlockArrayDraftTokenLocation(
        bufferCast<SizeType32>(seqAcceptedDraftTokenOffsets), bufferCast<SizeType32>(packedAcceptedDraftTokensIndices),
        bufferCast<SizeType32>(pastKeyValueLengths), pointerArray, offsetArray, layerCount, seqCount, numKVHeads,
        sizeInBytesPerKVHead, rewindDraftTokenCommonCount, rewindDraftTokenSeparateAdjustments,
        bufferCast<SizeType32>(seqSlotRemapping), bufferCast<SizeType32>(batchSlots), maxKVCacheLen, maxBlocksPerSeq,
        tokensPerBlock, canUseOneMoreBlock, stream);
}

} // namespace tensorrt_llm::runtime::kernels
