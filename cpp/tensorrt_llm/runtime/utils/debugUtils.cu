#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#ifndef CUDART_VERSION
#error CUDART_VERSION Undefined!
#elif (CUDART_VERSION >= 11050)
#include <hipcub/hipcub.hpp>
#else
#include "3rdparty/hipcub/hipcub.hpp"
#endif

#include "tensorrt_llm/runtime/utils/debugUtils.h"

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include <cfloat>
#include <string>

namespace
{
template <typename T, int blockSize>
__global__ void checkTensorInvalidKernel(T const* data, std::size_t size, int* foundInvalid)
{
    auto tidx = blockIdx.x * blockDim.x + threadIdx.x;

    int32_t found = 0;

    for (auto idx = tidx; idx < size; idx += blockDim.x * gridDim.x)
    {
        auto value = static_cast<float>(data[idx]);
        if (isnan(value) || isinf(value))
        {
            found = 1;
            break;
        }
    }

    typedef hipcub::BlockReduce<int32_t, blockSize> BlockReduceT;

    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduceT::TempStorage tempStorage;

    // Compute block-wide maximum
    int blockFound = BlockReduceT(tempStorage).Reduce(found, hipcub::Max());

    // Have thread 0 write out block's result
    if (threadIdx.x == 0)
    {
        atomicCAS(foundInvalid, 0, blockFound);
    }
}

__global__ void stallStreamKernel(int const microSeconds)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 700)
    for (int i = 0; i < microSeconds; ++i)
    {
        __nanosleep(1000);
    }
#endif
}
} // namespace

using namespace tensorrt_llm::runtime;
namespace tc = tensorrt_llm::common;

namespace tensorrt_llm::runtime::utils
{

template <typename T>
void invokeCheckTensorInvalidKernel(T const* data, std::size_t size, int* foundInvalid, hipStream_t stream)
{
    constexpr uint32_t kThreadsPerCta = 256;
    checkTensorInvalidKernel<T, kThreadsPerCta>
        <<<tc::ceilDiv(size, kThreadsPerCta), kThreadsPerCta, 0, stream>>>(data, size, foundInvalid);
}

template void invokeCheckTensorInvalidKernel(
    float const* data, std::size_t size, int* foundInvalid, hipStream_t stream);
template void invokeCheckTensorInvalidKernel(
    half const* data, std::size_t size, int* foundInvalid, hipStream_t stream);
template void invokeCheckTensorInvalidKernel(
    __hip_bfloat16 const* data, std::size_t size, int* foundInvalid, hipStream_t stream);
template void invokeCheckTensorInvalidKernel(
    __hip_fp8_e4m3_fnuz const* data, std::size_t size, int* foundInvalid, hipStream_t stream);

template <typename T>
void printLogitsKeyInfo(ITensor const& tensor, std::string const& infoStr)
{
    auto const& shape = tensor.getShape();
    auto const volume = ITensor::volume(shape);

    BufferManager::ITensorPtr host{};
    T const* hostData;
    if (tensor.getMemoryType() == MemoryType::kGPU)
    {
        auto streamPtr = std::make_shared<CudaStream>();
        BufferManager manager{streamPtr};
        host = manager.copyFrom(tensor, MemoryType::kCPU);
        streamPtr->synchronize();
        hostData = bufferCast<T>(*host);
    }
    else
    {
        hostData = bufferCast<T>(tensor);
    }

    std::stringstream ss;
    ss << infoStr;
    ss << " Shape: " << shape;
    ss << "; Top 5: ";
    for (size_t ki = 0; ki < 5; ++ki)
    {
        ss << static_cast<float>(hostData[ki]) << ", ";
    }

    ss << " Last 5: ";
    for (size_t ki = volume - 6; ki < volume; ++ki)
    {
        ss << static_cast<float>(hostData[ki]) << ", ";
    }

    // find max, min, avg
    double mSum = 0.f;
    float mMax = -FLT_MAX;
    float mMin = FLT_MAX;

    for (size_t ki = 0; ki < volume; ++ki)
    {
        float value = static_cast<float>(hostData[ki]);
        mSum += value;
        if (value > mMax)
        {
            mMax = value;
        }
        if (value < mMin)
        {
            mMin = value;
        }
    }
    float mAvg = mSum / volume;

    ss << " avg: " << mAvg << ", min: " << mMin << ", max: " << mMax << std::endl;

    TLLM_LOG_TRACE(ss.str());
}

template void printLogitsKeyInfo<float>(ITensor const& tensor, std::string const& infoStr);
template void printLogitsKeyInfo<half>(ITensor const& tensor, std::string const& infoStr);
template void printLogitsKeyInfo<__hip_bfloat16>(ITensor const& tensor, std::string const& infoStr);
template void printLogitsKeyInfo<__hip_fp8_e4m3_fnuz>(ITensor const& tensor, std::string const& infoStr);

template <typename T>
bool tensorHasInvalid(ITensor const& tensor, BufferManager const& manager, std::string const& infoStr)
{
    printLogitsKeyInfo<T>(tensor, infoStr);
    auto foundInvalid = BufferManager::pinnedPool(ITensor::makeShape({1}), nvinfer1::DataType::kINT32);
    auto foundInvalidPtr = bufferCast<int32_t>(*foundInvalid);
    foundInvalidPtr[0] = 0;
    auto const size = tensor.getSize();
    invokeCheckTensorInvalidKernel(bufferCast<T>(tensor), size, foundInvalidPtr, manager.getStream().get());
    manager.getStream().synchronize();
    return static_cast<bool>(foundInvalidPtr[0]);
}

template bool tensorHasInvalid<float>(ITensor const& tensor, BufferManager const& manager, std::string const& infoStr);
template bool tensorHasInvalid<half>(ITensor const& tensor, BufferManager const& manager, std::string const& infoStr);
template bool tensorHasInvalid<__hip_bfloat16>(
    ITensor const& tensor, BufferManager const& manager, std::string const& infoStr);
template bool tensorHasInvalid<__hip_fp8_e4m3_fnuz>(
    ITensor const& tensor, BufferManager const& manager, std::string const& infoStr);

bool tensorHasInvalid(
    size_t M, size_t K, nvinfer1::DataType type, void const* data, hipStream_t stream, std::string const& infoStr)
{
    auto tensorView = ITensor::wrap(
        const_cast<void*>(data), type, ITensor::makeShape({static_cast<int32_t>(M), static_cast<int32_t>(K)}));
    auto manager = BufferManager(std::make_shared<CudaStream>(stream));
    if (type == nvinfer1::DataType::kFLOAT)
    {
        return tensorHasInvalid<float>(*tensorView, manager, infoStr);
    }
    else if (type == nvinfer1::DataType::kHALF)
    {
        return tensorHasInvalid<half>(*tensorView, manager, infoStr);
    }
    else if (type == nvinfer1::DataType::kBF16)
    {
        return tensorHasInvalid<__hip_bfloat16>(*tensorView, manager, infoStr);
    }
    else if (type == nvinfer1::DataType::kFP8)
    {
        return tensorHasInvalid<__hip_fp8_e4m3_fnuz>(*tensorView, manager, infoStr);
    }
    else
    {
        TLLM_THROW("Not supported type for Nan check");
    }
}

int stallStream(char const* name, std::optional<hipStream_t> stream, std::optional<int> delay)
{
    int delay_val = 0;
    if (delay)
    {
        delay_val = delay.value();
    }
    else
    {
        char const* const env = std::getenv(name);
        if (env != nullptr)
        {
            delay_val = std::stoi(env);
        }
    }
    if (stream && delay_val > 0)
    {
        stallStreamKernel<<<1, 32, 0, stream.value()>>>(delay_val);
    }
    return delay_val;
}

} // namespace tensorrt_llm::runtime::utils
