#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 2025 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "cacheSplitConcat.h"
#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaFp8Utils.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/executor/dataTransceiverState.h"
#include "tensorrt_llm/executor/tensor.h"
#include "tensorrt_llm/executor/types.h"
#include "tensorrt_llm/runtime/bufferManager.h"
#include "tensorrt_llm/runtime/iBuffer.h"
#include "tensorrt_llm/runtime/iTensor.h"
#include "tensorrt_llm/runtime/utils/mpiUtils.h"
#include <NvInferRuntimeBase.h>
#include <cstddef>
#include <cstdint>
#include <sstream>
#include <string>
#include <vector>

namespace tensorrt_llm::executor::kv_cache
{

namespace
{
inline bool isPowerOfTwo(int n)
{
    return n > 0 && (n & (n - 1)) == 0;
}
} // namespace

// inputBlockNums: [outputBlockNum, inputRanks.size]
// [PP, TP]
TargetRanksInfo TargetRanksInfoForDP(
    kv_cache::CacheState const& peerCacheState, kv_cache::CacheState const& selfCacheState, int selfRank)
{
    auto const& peerParConfig = peerCacheState.getParallelConfig();
    auto const& selfParConfig = selfCacheState.getParallelConfig();

    auto const peerPPNum = peerParConfig.mPipelineParallelism;
    auto const selfPPNum = selfParConfig.mPipelineParallelism;
    auto const peerTPNum = peerParConfig.mTensorParallelism;
    auto const selfTPNum = selfParConfig.mTensorParallelism;

    for (auto val : {peerPPNum, selfPPNum, peerTPNum, selfTPNum})
    {
        TLLM_CHECK(isPowerOfTwo(val));
    }

    auto const selfTPRank = selfRank % selfParConfig.mTensorParallelism;
    auto const selfPPRank = selfRank / selfParConfig.mTensorParallelism;

    int peerPPRankStart = 0;
    int mDomainPPSize = 1;
    int peerPPRankEnd = 0;

    if (selfPPNum <= peerPPNum)
    {
        mDomainPPSize = peerPPNum / selfPPNum;
        peerPPRankStart = selfPPRank * mDomainPPSize;
        peerPPRankEnd = (selfPPRank + 1) * mDomainPPSize;
    }
    else
    {
        peerPPRankStart = selfPPRank / (selfPPNum / peerPPNum);
        peerPPRankEnd = peerPPRankStart + mDomainPPSize;
    }

    int peerTPRankStart = 0;
    int mDomainTPSize = 1;
    int peerTPRankEnd = 0;

    int const peerDpRank = peerParConfig.mEnableAttentionDP ? peerParConfig.mDPrank : 0;
    int const selfTPSizePerDPGroup = selfParConfig.mEnableAttentionDP ? selfTPNum / selfParConfig.mDPsize : selfTPNum;
    int const peerTPSizePerDPGroup = peerParConfig.mEnableAttentionDP ? peerTPNum / peerParConfig.mDPsize : peerTPNum;

    int const selfNbHeadsPerLayer = selfCacheState.getModelConfig().mNbKvHeadsPerLayer[0];
    int const peerNbHeadsPerLayer = peerCacheState.getModelConfig().mNbKvHeadsPerLayer[0];
    int const selfTPrankInDPGroup = selfTPRank % selfTPSizePerDPGroup;

    if (selfTPSizePerDPGroup <= peerTPSizePerDPGroup)
    {
        mDomainTPSize = peerTPSizePerDPGroup / selfTPSizePerDPGroup;
        peerTPRankStart = selfTPrankInDPGroup * mDomainTPSize + peerDpRank * peerTPSizePerDPGroup;
        peerTPRankEnd = peerTPRankStart + mDomainTPSize;
    }
    else
    {
        peerTPRankStart
            = selfTPrankInDPGroup / (selfTPSizePerDPGroup / peerTPSizePerDPGroup) + peerDpRank * peerTPSizePerDPGroup;
        peerTPRankEnd = peerTPRankStart + mDomainTPSize;
    }

    std::vector<int> retRanks;
    for (int i = peerTPRankStart; i < peerTPRankEnd; i++)
    {
        for (int j = peerPPRankStart; j < peerPPRankEnd; j++)
        {
            int irank = j * peerTPNum + i;
            retRanks.push_back(irank);
        }
    }

    int mDupHeadFactor = 1;
    int mPeerDupHeadFactor = 1;

    if (selfNbHeadsPerLayer * selfTPSizePerDPGroup > peerNbHeadsPerLayer * peerTPSizePerDPGroup)
    {
        mDupHeadFactor = (selfNbHeadsPerLayer * selfTPSizePerDPGroup) / (peerNbHeadsPerLayer * peerTPSizePerDPGroup);
    }
    if (peerNbHeadsPerLayer * peerTPSizePerDPGroup > selfNbHeadsPerLayer * selfTPSizePerDPGroup)
    {
        mPeerDupHeadFactor
            = (peerNbHeadsPerLayer * peerTPSizePerDPGroup) / (selfNbHeadsPerLayer * selfTPSizePerDPGroup);
    }

    return {mDomainPPSize, mDomainTPSize, std::move(retRanks), mDupHeadFactor, mPeerDupHeadFactor};
}

TargetRanksInfo targetIRanks(
    kv_cache::CacheState const& peerCacheState, kv_cache::CacheState const& selfCacheState, int selfRank)
{
    return TargetRanksInfoForDP(peerCacheState, selfCacheState, selfRank);
}

template <typename T>
struct BlockInfo
{
    T* data;
    int startTokenId;
    int tokensPerBlock;
    int startHeadId;
    int headsPerBlock;
    int startLayerId;
    int layersPerBlock;
    int dimsPerHead;
    size_t offset; // (data-offset)[idx]

    __forceinline__ __device__ __host__ T* getKblockPtr(int layerid) const
    {
        return data + (layerid * 2) * headsPerBlock * tokensPerBlock * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T* getVblockPtr(int layerid) const
    {
        return data + (layerid * 2 + 1) * headsPerBlock * tokensPerBlock * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T* getKDimsPtr(int layerid, int headid, int tokenid)
    {
        return getKblockPtr(layerid) + headid * tokensPerBlock * dimsPerHead + tokenid * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T const* getKDimsPtr(int layerid, int headid, int tokenid) const
    {
        return getKblockPtr(layerid) + headid * tokensPerBlock * dimsPerHead + tokenid * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T* getVDimsPtr(int layerid, int headid, int tokenid)
    {
        return getVblockPtr(layerid) + headid * tokensPerBlock * dimsPerHead + tokenid * dimsPerHead;
    }

    __forceinline__ __device__ __host__ T const* getVDimsPtr(int layerid, int headid, int tokenid) const
    {
        return getVblockPtr(layerid) + headid * tokensPerBlock * dimsPerHead + tokenid * dimsPerHead;
    }

    std::string to_string() const
    {
        std::stringstream ss;
        ss << "{data ptr: " << data << ", startTokenId: " << startTokenId << ", tokensPerBlock: " << tokensPerBlock
           << ", startHeadId: " << startHeadId << ", headsPerBlock: " << headsPerBlock
           << ", startLayerId:" << startLayerId << ", layersPerBlock: " << layersPerBlock
           << ", dimsPerHead: " << dimsPerHead << ", offset: " << offset << "}";
        return ss.str();
    }
};

// Reference to blockPtr

// Block shape: [numHeads, numTokens, dimsPerHead]
// CacheBlock shape: [numLayers, 2, mBlockSize]
// Note: mBlockSize refers to the size of each block

// Handling key and value copying
// Note: k and v are not stored contiguously in memory

__forceinline__ __device__ int getInputBlockId(int outputBlockId, int headId, int layerId, int inputBlockNumEachOutput,
    int headNumPerBlock, int layerNumPerBlock, int headNumInputModel, int layerNumInputModel)
{
    int const offset = outputBlockId * inputBlockNumEachOutput;
    int const layerOffset = layerId / layerNumPerBlock;
    int const headOffset = headId / headNumPerBlock;
    int const headBlockNum = headNumInputModel / headNumPerBlock;
    return offset + layerOffset * headBlockNum + headOffset;
}

// subWarpSize * subWarpGroupSize
template <typename T, int subWarpSize, int subWarpNumInGroup, int vecSizeByte>
__global__ void splitAndConcatBlocksKernel(BlockInfo<T> const* iBlockInfo, BlockInfo<T>* oBlockInfo, int iBlockNum,
    int iNumBlockEachO, int oBlockNum, int headNumInputModel, int layerNumInputModel, int iHeadsPerBlock,
    int iLayersPerBlock)
{
    // blockDim.y corresponds to the number of output blocks
    // blockDim.x corresponds to the number of layers

    // Warp-level parallelism spans heads * tokens
    // Thread-level parallelism spans dimsPerHead

    // input_id can be derived from output_id, layer_id, and head_id
    // Total number of CUDA blocks = numLayers * outputBlockNum

    int const subWarpId = threadIdx.x / subWarpSize;
    int const laneId = threadIdx.x % subWarpSize;
    int const subWarpNum = blockDim.x / subWarpSize;
    int const subWarpGroupId = subWarpId / subWarpNumInGroup; //
    int const subWarpGroupNum = subWarpNum / subWarpNumInGroup;
    int const subWarpIdInGroup = subWarpId % subWarpNumInGroup;
    static_assert(vecSizeByte >= sizeof(T));
    int constexpr numElePerThread = vecSizeByte / sizeof(T);
    using VecType = typename common::BytesToType<vecSizeByte>::type;

#pragma unroll 1
    for (int oBlockId = blockIdx.y; oBlockId < oBlockNum; oBlockId += gridDim.y)
    {
        int oLayerNum = oBlockInfo[oBlockId].layersPerBlock;
        int headNum = oBlockInfo[oBlockId].headsPerBlock;
        int tokenNum = oBlockInfo[oBlockId].tokensPerBlock;
        int dimsPerHead = oBlockInfo[oBlockId].dimsPerHead;
#pragma unroll 1

        for (int layerid = blockIdx.x; layerid < oLayerNum; layerid += gridDim.x)
        {
#pragma unroll 1
            for (int headId = subWarpGroupId; headId < headNum; headId += subWarpGroupNum)
            {
                int const targetHeadId = oBlockInfo[oBlockId].startHeadId + headId;
                int const targetLayerId = oBlockInfo[oBlockId].startLayerId + layerid;

                int const iBlockId = getInputBlockId(oBlockId, targetHeadId, targetLayerId, iNumBlockEachO,
                    iHeadsPerBlock, iLayersPerBlock, headNumInputModel, layerNumInputModel);
                int const iLayerId = targetLayerId % iLayersPerBlock;
                int const iHeadId = targetHeadId % iHeadsPerBlock;
#pragma unroll 1
                for (int tokenId = subWarpIdInGroup; tokenId < tokenNum; tokenId += subWarpNumInGroup)
                {
                    T* oKPtr = oBlockInfo[oBlockId].getKDimsPtr(layerid, headId, tokenId);
                    T const* iKPtr = iBlockInfo[iBlockId].getKDimsPtr(iLayerId, iHeadId, tokenId);
                    T* oVPtr = oBlockInfo[oBlockId].getVDimsPtr(layerid, headId, tokenId);
                    T const* iVPtr = iBlockInfo[iBlockId].getVDimsPtr(iLayerId, iHeadId, tokenId);
#pragma unroll 1
                    for (int channelId = laneId * numElePerThread; channelId < dimsPerHead;
                         channelId += (subWarpSize * numElePerThread))
                    {
                        common::copy<vecSizeByte>(iKPtr + channelId, oKPtr + channelId);
                        common::copy<vecSizeByte>(iVPtr + channelId, oVPtr + channelId);
                    }
                }
            }
        }
    }
}

template <typename T>
void concatKVCache(runtime::ITensor::SharedPtr* inputBlocks, int inputBlockNum, std::vector<int> const& inputRanks,
    kv_cache::CacheState const& iCacheState, runtime::ITensor::SharedPtr* outputBlocks, int outputBlockNum, int oRank,
    kv_cache::CacheState const& oCacheState, runtime::BufferManager const& bufferManager)

{
    TLLM_CHECK_WITH_INFO(!inputRanks.empty(), "inputRanks should not be empty.");
    TLLM_CHECK_WITH_INFO(inputBlockNum == outputBlockNum * inputRanks.size(),
        "inputBlockNum must equal outputBlockNum multiplied by the size of inputRanks.");
    TLLM_CHECK(inputRanks == targetIRanks(iCacheState, oCacheState, oRank).mIRanks);

    auto const& iParallelConfig = iCacheState.getParallelConfig();
    auto const& oParallelConfig = oCacheState.getParallelConfig();
    auto const& iModelConfig = iCacheState.getModelConfig();
    auto const& oModelConfig = oCacheState.getModelConfig();

    int const inputAllRankNum = iParallelConfig.mPipelineParallelism * iParallelConfig.mTensorParallelism;
    std::vector<BlockInfo<T>> blockInfos(outputBlockNum * inputAllRankNum + outputBlockNum);

    auto fillBlockInfo = [](kv_cache::CacheState const& cacheState, runtime::ITensor::SharedPtr buffer, int rank)
    {
        const auto& parallelConfig = cacheState.getParallelConfig();
        const auto& modelConfig = cacheState.getModelConfig();

        const int tpRank = rank % parallelConfig.mTensorParallelism;
        const int ppRank = rank / parallelConfig.mTensorParallelism;
        const int ppNum = parallelConfig.mPipelineParallelism;
        const int headsPerBlock = modelConfig.mNbKvHeadsPerLayer[0];
        const int layersPerBlock = modelConfig.mNbKvHeadsPerLayer.size() / ppNum;

        const int tokensPerBlock = modelConfig.mTokensPerBlock;
        const int dimsPerBlock = modelConfig.mSizePerHead;
        const int startHead = tpRank * headsPerBlock;
        const int startLayer = ppRank * layersPerBlock;

        constexpr int startTokenId = 0;
        auto* data = static_cast<T*>(buffer->data());
        return BlockInfo<T>{
            data, startTokenId, tokensPerBlock, startHead, headsPerBlock, startLayer, layersPerBlock, dimsPerBlock, 0};
    };
    // fill blcokInfo from CacheState and inputBlocks
    for (int oi = 0; oi < outputBlockNum; oi++)
    {
        int iRankNum = inputRanks.size();
        for (int i = 0; i < iRankNum; i++)
        {
            int iRank = inputRanks[i];
            blockInfos[oi * inputAllRankNum + iRank]
                = fillBlockInfo(iCacheState, inputBlocks[oi * iRankNum + i], iRank);
        }

        blockInfos[outputBlockNum * inputAllRankNum + oi] = fillBlockInfo(oCacheState, outputBlocks[oi], oRank);
    }
    runtime::BufferManager::IBufferPtr blockInfosDeviceBuffer
        = bufferManager.gpu(sizeof(BlockInfo<T>) * (blockInfos.size()), nvinfer1::DataType::kUINT8);
    bufferManager.copy((blockInfos.data()), *blockInfosDeviceBuffer, runtime::MemoryType::kCPU);

    BlockInfo<T>* iBlockInfoDevice = static_cast<BlockInfo<T>*>(blockInfosDeviceBuffer->data());
    BlockInfo<T>* oBlockInfoDevice = iBlockInfoDevice + outputBlockNum * inputAllRankNum;

    constexpr int subWarpSize = 8;
    constexpr int subWarpNumInGroup = 8;
    constexpr int blockDimx = 128;

    int oPPNum = oParallelConfig.mPipelineParallelism;
    int iPPNum = iParallelConfig.mPipelineParallelism;
    unsigned int gridDimx = oModelConfig.mNbKvHeadsPerLayer.size() / oPPNum;
    unsigned int gridDimy = outputBlockNum;

    dim3 gridDim{gridDimx, gridDimy};
    int const headsInputModel = iModelConfig.mNbKvHeadsPerLayer[0] * iParallelConfig.mTensorParallelism;
    int const layersInputModel = iModelConfig.mNbKvHeadsPerLayer.size();
    int const iHeadsPerBlock = iModelConfig.mNbKvHeadsPerLayer[0];
    int const iLayersPerBlock = iModelConfig.mNbKvHeadsPerLayer.size() / iPPNum;
    int const sizePerHead = oModelConfig.mSizePerHead;
    int const remainder = sizePerHead * sizeof(T) % 16;
    switch (remainder)
    {
    case 0:
    {
        splitAndConcatBlocksKernel<T, subWarpSize, subWarpNumInGroup, 16>
            <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel, layersInputModel,
                iHeadsPerBlock, iLayersPerBlock);
        break;
    }
    case 8:
    {
        splitAndConcatBlocksKernel<T, subWarpSize, subWarpNumInGroup, 8>
            <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel, layersInputModel,
                iHeadsPerBlock, iLayersPerBlock);
        break;
    }
    case 4:
    case 12:
    {
        if constexpr (sizeof(T) <= 4)
        {
            splitAndConcatBlocksKernel<T, subWarpSize, subWarpNumInGroup, 4>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                    outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel,
                    layersInputModel, iHeadsPerBlock, iLayersPerBlock);
            break;
        }
    }
    case 2:
    case 6:
    case 10:
    case 14:
    {
        if constexpr (sizeof(T) <= 2)
        {

            splitAndConcatBlocksKernel<T, subWarpSize, subWarpNumInGroup, 2>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                    outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel,
                    layersInputModel, iHeadsPerBlock, iLayersPerBlock);
            break;
        }
    }
    default:
    {
        if constexpr (sizeof(T) <= 1)
        {
            splitAndConcatBlocksKernel<T, subWarpSize, subWarpNumInGroup, 1>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(iBlockInfoDevice, oBlockInfoDevice,
                    outputBlockNum * inputAllRankNum, inputAllRankNum, outputBlockNum, headsInputModel,
                    layersInputModel, iHeadsPerBlock, iLayersPerBlock);
        }
        else
        {
            TLLM_THROW("concatKVCacheDispatch encountered an unsupported data type error.");
        }
    }
    }
}

void concatKVCacheDispatch(runtime::ITensor::SharedPtr* inputBlocks, int inputBlockNum,
    std::vector<int> const& inputRanks, kv_cache::CacheState const& iCacheState,
    runtime::ITensor::SharedPtr* outputBlocks, int outputBlockNum, int oRanks, kv_cache::CacheState const& oCacheState,
    runtime::BufferManager const& bufferManager)
{
    auto dataType = outputBlocks[0]->getDataType();
    int dataSize = tensorrt_llm::common::getDTypeSize(dataType);

    switch (dataSize)
    {
    case 8:
    {
        concatKVCache<int64_t>(inputBlocks, inputBlockNum, inputRanks, iCacheState, outputBlocks, outputBlockNum,
            oRanks, oCacheState, bufferManager);
        break;
    }
    case 4:
    {
        concatKVCache<int32_t>(inputBlocks, inputBlockNum, inputRanks, iCacheState, outputBlocks, outputBlockNum,
            oRanks, oCacheState, bufferManager);
        break;
    }
    case 2:
    {
        concatKVCache<int16_t>(inputBlocks, inputBlockNum, inputRanks, iCacheState, outputBlocks, outputBlockNum,
            oRanks, oCacheState, bufferManager);
        break;
    }

    case 1:
    {
        concatKVCache<int8_t>(inputBlocks, inputBlockNum, inputRanks, iCacheState, outputBlocks, outputBlockNum, oRanks,
            oCacheState, bufferManager);
        break;
    }

    default:
    {
        TLLM_THROW("concatKVCacheDispatch encountered an unsupported data type error.");
    }
    }
}

nvinfer1::Dims makeShapeFromCacheState(kv_cache::CacheState const& cacheState)
{

    int64_t blockSize = static_cast<int64_t>(cacheState.getModelConfig().mNbKvHeadsPerLayer[0]
        * cacheState.getModelConfig().mTokensPerBlock * cacheState.getModelConfig().mSizePerHead);
    int PPNum = cacheState.getParallelConfig().mPipelineParallelism;
    return runtime::ITensor::makeShape(
        {static_cast<int64_t>(cacheState.getModelConfig().mNbKvHeadsPerLayer.size() / PPNum),
            cacheState.getAttentionConfig().mKvFactor, blockSize});
}

// MLA Head 1: One thread block per [(2), tokens, dimsPerHead]

template <typename T, int subWarpSize, int vecSizeByte>
__global__ void splitKVCacheForMLAKernel(T const** __restrict__ inputBlocks, T** __restrict__ outputCaches,
    int tokensPerBlock, int numLayers, int headNum, int dimsPerHead, int inputBlockNum, int DomainPPSize,
    int DomainTPSize, int layerNumDomainPP, int kvFactor)
{
    int const subWarpId = threadIdx.x / subWarpSize;
    int const laneId = threadIdx.x % subWarpSize;
    int const subWarpNum = blockDim.x / subWarpSize;

    static_assert(vecSizeByte >= sizeof(T));
    int constexpr numElePerThread = vecSizeByte / sizeof(T);
    using VecType = typename common::BytesToType<vecSizeByte>::type;
#pragma unroll 1

    for (int blockId = blockIdx.y; blockId < inputBlockNum; blockId += gridDim.y)
    {
#pragma unroll 1

        for (int layerId = blockIdx.x; layerId < numLayers; layerId += gridDim.x)
        {
#pragma unroll 1
            for (int headId = 0; headId < headNum; headId++)
            {
                T const* inputBlockPtr = inputBlocks[blockId];
                T const* kInputPtr = inputBlockPtr + layerId * kvFactor * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;
                int const outputCacheIdx = layerId / layerNumDomainPP;
                T* outputCachePtr = outputCaches[outputCacheIdx];
                int const layerIdInDomainPP = layerId % layerNumDomainPP;
                int const headIdInDomainTP = headId;

                T* kOutputPtr = outputCachePtr
                    + blockId * (layerNumDomainPP * kvFactor * headNum * tokensPerBlock * dimsPerHead)
                    + layerIdInDomainPP * kvFactor * headNum * tokensPerBlock * dimsPerHead
                    + headIdInDomainTP * tokensPerBlock * dimsPerHead;
                int const kvOffset = headNum * tokensPerBlock * dimsPerHead;
#pragma unroll 1
                for (int tokenId = subWarpId; tokenId < tokensPerBlock; tokenId += subWarpNum)
                {
                    T const* iKPtr = kInputPtr + tokenId * dimsPerHead;
                    T* oKPtr = kOutputPtr + tokenId * dimsPerHead;
#pragma unroll 1
                    for (int channelId = laneId * numElePerThread; channelId < dimsPerHead;
                         channelId += subWarpSize * numElePerThread)
                    {
#pragma unroll 1
                        for (int kvId = 0; kvId < kvFactor; kvId++)
                        {
                            common::copy<vecSizeByte>(
                                iKPtr + kvId * kvOffset + channelId, oKPtr + kvId * kvOffset + channelId);
                        }
                    }
                }
            }
        }
    }
}

// Block shape: [head, tokens, dimsPerHead]
// CacheBlock shape: [numLayers, 2, mBlockSize]
// Output split caches shape: [outputSplitCaches, numLayers, 2, head, tokensPerBlock, dimsPerHead]
// Note: The number of tokens can be large
// subWarpSize * subWarpGroupSize

template <typename T, int subWarpSize, int subWarpNumInGroup, int vecSizeByte>
__global__ void splitKVCacheKernel(T const** __restrict__ inputBlocks, T** __restrict__ outputCaches,
    int tokensPerBlock, int numLayers, int headNum, int dimsPerHead, int inputBlockNum, int DomainPPSize,
    int DomainTPSize, int layerNumDomainPP, int headNumDomainTP)
{

    int const subWarpId = threadIdx.x / subWarpSize;
    int const laneId = threadIdx.x % subWarpSize;
    int const subWarpNum = blockDim.x / subWarpSize;
    int const subWarpGroupId = subWarpId / subWarpNumInGroup; //
    int const subWarpGroupNum = subWarpNum / subWarpNumInGroup;
    int const subWarpIdInGroup = subWarpId % subWarpNumInGroup;
    static_assert(vecSizeByte >= sizeof(T));
    int constexpr numElePerThread = vecSizeByte / sizeof(T);
    using VecType = typename common::BytesToType<vecSizeByte>::type;
#pragma unroll 1

    for (int blockId = blockIdx.y; blockId < inputBlockNum; blockId += gridDim.y)
    {
#pragma unroll 1

        for (int layerId = blockIdx.x; layerId < numLayers; layerId += gridDim.x)
        {
#pragma unroll 1

            for (int headId = subWarpGroupId; headId < headNum; headId += subWarpGroupNum)
            {

                T const* inputBlockPtr = inputBlocks[blockId];
                T const* kInputPtr = inputBlockPtr + layerId * 2 * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;
                T const* vInputPtr = inputBlockPtr + (layerId * 2 + 1) * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;

                int outputCacheIdx = headId / headNumDomainTP * DomainPPSize + layerId / layerNumDomainPP;
                T* outputCachePtr = outputCaches[outputCacheIdx];
                int layerIdInDomainPP = layerId % layerNumDomainPP;

                int headIdInDomainTP = headId % headNumDomainTP;
                T* kOutputPtr = outputCachePtr
                    + blockId * (layerNumDomainPP * 2 * headNumDomainTP * tokensPerBlock * dimsPerHead)
                    + layerIdInDomainPP * 2 * headNumDomainTP * tokensPerBlock * dimsPerHead
                    + headIdInDomainTP * tokensPerBlock * dimsPerHead;

                T* vOutputPtr = kOutputPtr + headNumDomainTP * tokensPerBlock * dimsPerHead;
#pragma unroll 1

                for (int tokenId = subWarpIdInGroup; tokenId < tokensPerBlock; tokenId += subWarpNumInGroup)
                {
                    auto baseOffset = tokenId * dimsPerHead;
#pragma unroll 1
                    for (int channelId = laneId * numElePerThread; channelId < dimsPerHead;
                         channelId += (subWarpSize * numElePerThread))
                    {
                        auto offset = baseOffset + channelId;
                        common::copy<vecSizeByte>(kInputPtr + offset, kOutputPtr + offset);
                        common::copy<vecSizeByte>(vInputPtr + offset, vOutputPtr + offset);
                    }
                }
            }
        }
    }
}

template <typename T, int subWarpSize, int vecSizeByte>
__global__ void concatKVCacheForMLAKernel(T const** __restrict__ inputCaches, T** __restrict__ outputBlocks,
    int tokensPerBlock, int numLayers, int headNum, int dimsPerHead, int outputBlockNum, int DomainPPSize,
    int DomainTPSize, int layerNumDomainPP, int kvFactor)
{

    int const subWarpId = threadIdx.x / subWarpSize;
    int const laneId = threadIdx.x % subWarpSize;
    int const subWarpNum = blockDim.x / subWarpSize;
    static_assert(vecSizeByte >= sizeof(T));
    int constexpr numElePerThread = vecSizeByte / sizeof(T);
    using VecType = typename common::BytesToType<vecSizeByte>::type;
#pragma unroll 1
    for (int blockId = blockIdx.y; blockId < outputBlockNum; blockId += gridDim.y)
    {
#pragma unroll 1
        for (int layerId = blockIdx.x; layerId < numLayers; layerId += gridDim.x)
        {

#pragma unroll 1

            for (int headId = 0; headId < headNum; headId++)
            {
                T* outputBlockPtr = outputBlocks[blockId];
                T* kOutputPtr = outputBlockPtr + layerId * kvFactor * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;
                int inputCacheIdx = layerId / layerNumDomainPP;
                T const* inputCachePtr = inputCaches[inputCacheIdx];
                int layerIdInDomainPP = layerId % layerNumDomainPP;
                int headIdInDomainTP = headId;

                T const* kInputPtr = inputCachePtr
                    + blockId * (layerNumDomainPP * kvFactor * headNum * tokensPerBlock * dimsPerHead)
                    + layerIdInDomainPP * kvFactor * headNum * tokensPerBlock * dimsPerHead
                    + headIdInDomainTP * tokensPerBlock * dimsPerHead;
                int const kvOffset = headNum * tokensPerBlock * dimsPerHead;
#pragma unroll 1
                for (int tokenId = subWarpId; tokenId < tokensPerBlock; tokenId += subWarpNum)
                {
                    T const* iKPtr = kInputPtr + tokenId * dimsPerHead;
                    T* oKPtr = kOutputPtr + tokenId * dimsPerHead;
#pragma unroll 1
                    for (int channelId = laneId * numElePerThread; channelId < dimsPerHead;
                         channelId += subWarpSize * numElePerThread)
                    {
#pragma unroll 1
                        for (int kvId = 0; kvId < kvFactor; kvId++)
                        {
                            common::copy<vecSizeByte>(
                                iKPtr + kvId * kvOffset + channelId, oKPtr + kvId * kvOffset + channelId);
                        }
                    }
                }
            }
        }
    }
}

template <typename T, int subWarpSize, int subWarpNumInGroup, int vecSizeByte>
__global__ void concatKVCacheKernel(T const** __restrict__ inputCaches, T** __restrict__ outputBlocks,
    int tokensPerBlock, int numLayers, int headNum, int dimsPerHead, int outputBlockNum, int DomainPPSize,
    int DomainTPSize, int layerNumDomainPP, int headNumDomainTP)
{
    int const subWarpId = threadIdx.x / subWarpSize;
    int const laneId = threadIdx.x % subWarpSize;
    int const subWarpNum = blockDim.x / subWarpSize;
    int const subWarpGroupId = subWarpId / subWarpNumInGroup; //
    int const subWarpGroupNum = subWarpNum / subWarpNumInGroup;
    int const subWarpIdInGroup = subWarpId % subWarpNumInGroup;
    static_assert(vecSizeByte >= sizeof(T));
    int constexpr numElePerThread = vecSizeByte / sizeof(T);
    using VecType = typename common::BytesToType<vecSizeByte>::type;
#pragma unroll 1
    for (int blockId = blockIdx.y; blockId < outputBlockNum; blockId += gridDim.y)
    {
#pragma unroll 1
        for (int layerId = blockIdx.x; layerId < numLayers; layerId += gridDim.x)
        {

#pragma unroll 1
            for (int headId = subWarpGroupId; headId < headNum; headId += subWarpGroupNum)
            {

                T* outputBlockPtr = outputBlocks[blockId];
                T* kOutputPtr = outputBlockPtr + layerId * 2 * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;
                T* vOutputPtr = outputBlockPtr + (layerId * 2 + 1) * headNum * tokensPerBlock * dimsPerHead
                    + headId * tokensPerBlock * dimsPerHead;

                int inputCacheIdx = headId / headNumDomainTP * DomainPPSize + layerId / layerNumDomainPP;
                T const* inputCachePtr = inputCaches[inputCacheIdx];
                int layerIdInDomainPP = layerId % layerNumDomainPP;

                int headIdInDomainTP = headId % headNumDomainTP;
                T const* kInputPtr = inputCachePtr
                    + blockId * (layerNumDomainPP * 2 * headNumDomainTP * tokensPerBlock * dimsPerHead)
                    + layerIdInDomainPP * 2 * headNumDomainTP * tokensPerBlock * dimsPerHead
                    + headIdInDomainTP * tokensPerBlock * dimsPerHead;

                T const* vInputPtr = kInputPtr + headNumDomainTP * tokensPerBlock * dimsPerHead;
#pragma unroll 1
                for (int tokenId = subWarpIdInGroup; tokenId < tokensPerBlock; tokenId += subWarpNumInGroup)
                {
                    auto baseOffset = tokenId * dimsPerHead;
#pragma unroll 1
                    for (int channelId = laneId * numElePerThread; channelId < dimsPerHead;
                         channelId += (subWarpSize * numElePerThread))
                    {
                        auto offset = baseOffset + channelId;
                        common::copy<vecSizeByte>(kInputPtr + offset, kOutputPtr + offset);
                        common::copy<vecSizeByte>(vInputPtr + offset, vOutputPtr + offset);
                    }
                }
            }
        }
    }
}

template <typename T>
void splitKVCache(std::vector<runtime::ITensor::SharedPtr> const& kVCacheBlocks,
    std::vector<runtime::ITensor::SharedPtr>& outputSplitBlocks, kv_cache::CacheState const& destCacheState,
    kv_cache::CacheState const& selfCacheState, int selfIdx, runtime::BufferManager const& bufferManager)
{

    auto inputBlockNum = kVCacheBlocks.size();
    auto targetRankInfo = targetIRanks(destCacheState, selfCacheState, selfIdx);
    TLLM_CHECK(targetRankInfo.mIRanks.size()
        == (static_cast<size_t>(targetRankInfo.mDomainPPSize * targetRankInfo.mDomainTPSize)));
    auto outputCacheNum = targetRankInfo.mIRanks.size();
    if (selfCacheState.getAttentionConfig().mAttentionType == CacheState::AttentionType::kMLA)
    {
        outputCacheNum = targetRankInfo.mDomainPPSize;
    }
    else
    {
        outputCacheNum = outputCacheNum / targetRankInfo.mPeerDupHeadFactor;
    }
    TLLM_CHECK(outputCacheNum == outputSplitBlocks.size());
    TLLM_CHECK(inputBlockNum > 0);
    auto cacheBlockSize = kVCacheBlocks.at(0)->getSize();
    auto cacheDataType = kVCacheBlocks.at(0)->getDataType();
    std::vector<T*> cachePtrs;

    for (auto&& kvCacheBlock : kVCacheBlocks)
    {
        TLLM_CHECK(kvCacheBlock->getDataType() == cacheDataType);
        TLLM_CHECK(kvCacheBlock->getSize() == cacheBlockSize);
        cachePtrs.push_back(static_cast<T*>(kvCacheBlock->data()));
    }

    for (auto&& outputSplitBlock : outputSplitBlocks)
    {
        TLLM_CHECK(outputSplitBlock->getDataType() == cacheDataType);
        TLLM_CHECK(outputSplitBlock->getSize() == cacheBlockSize * inputBlockNum / outputCacheNum);
        cachePtrs.push_back(static_cast<T*>(outputSplitBlock->data()));
    }
    runtime::BufferManager::IBufferPtr PtrsDeviceBuffer
        = bufferManager.gpu(cachePtrs.size(), nvinfer1::DataType::kINT64);
    TLLM_CHECK(PtrsDeviceBuffer->getSizeInBytes() == cachePtrs.size() * sizeof(T*));
    bufferManager.copy(cachePtrs.data(), *PtrsDeviceBuffer, runtime::MemoryType::kCPU);

    constexpr int subWarpSize = 8;
    constexpr int subWarpNumInGroup = 8;
    constexpr int blockDimx = 128;

    auto const& selfParallelConfig = selfCacheState.getParallelConfig();
    auto const& selfModelConfig = selfCacheState.getModelConfig();
    auto const& destParallelConfig = destCacheState.getParallelConfig();
    auto const& selfAttentionConfig = selfCacheState.getAttentionConfig();

    int oPPNum = selfParallelConfig.mPipelineParallelism;

    // layers
    unsigned int gridDimx = selfModelConfig.mNbKvHeadsPerLayer.size() / oPPNum;
    // blockNum
    unsigned int gridDimy = inputBlockNum;

    dim3 gridDim{gridDimx, gridDimy};

    int const sizePerHead = selfModelConfig.mSizePerHead;
    T const** inputBlockPtrsDev = static_cast<T const**>(PtrsDeviceBuffer->data());
    T** outputCachePtrsDev = static_cast<T**>(PtrsDeviceBuffer->data()) + inputBlockNum;
    int const tokensPerBlock = selfModelConfig.mTokensPerBlock;
    int const numLayers = selfModelConfig.mNbKvHeadsPerLayer.size() / oPPNum;
    int const headNum = selfModelConfig.mNbKvHeadsPerLayer[0];
    int const dimsPerHead = selfModelConfig.mSizePerHead;
    int const DomainPPSize = targetRankInfo.mDomainPPSize;
    int const DomainTPSize = targetRankInfo.mDomainTPSize;
    int const layerNumDomainPP = numLayers / DomainPPSize;
    int const headNumDomainTP
        = headNum / (DomainTPSize / targetRankInfo.mPeerDupHeadFactor); // TODO: duplicate head factor
    int const kvFactor = selfAttentionConfig.mKvFactor;
    bool const isMLA = selfAttentionConfig.mAttentionType == CacheState::AttentionType::kMLA;
    constexpr int mlaSubWarpSize = 16;

    TLLM_LOG_DEBUG(
        "splitKVCache - numLayers: %d, headNum: %d, domainPPSize: %d, domainTPSize: %d, "
        "layersPerDomainPP: %d, headsPerDomainTP: %d",
        numLayers, headNum, DomainPPSize, DomainTPSize, layerNumDomainPP, headNumDomainTP);

    int const remainder = sizePerHead * sizeof(T) % 16;
    switch (remainder)
    {
    case 0:
    {
        if (isMLA)
        {
            splitKVCacheForMLAKernel<T, mlaSubWarpSize, 16><<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(
                inputBlockPtrsDev, outputCachePtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum,
                DomainPPSize, DomainTPSize, layerNumDomainPP, kvFactor);
        }
        else
        {
            splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 16>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);
        }
        break;
    }
    case 8:
    {
        if (isMLA)
        {
            splitKVCacheForMLAKernel<T, mlaSubWarpSize, 8><<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(
                inputBlockPtrsDev, outputCachePtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum,
                DomainPPSize, DomainTPSize, layerNumDomainPP, kvFactor);
        }
        else
        {
            splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 8>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);
        }
        break;
    }
    case 4:
    case 12:
    {
        if constexpr (sizeof(T) <= 4)
        {
            if (isMLA)
            {
                splitKVCacheForMLAKernel<T, mlaSubWarpSize, 4>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                        tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                        layerNumDomainPP, kvFactor);
            }
            else
            {
                splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 4>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                        tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                        layerNumDomainPP, headNumDomainTP);
            }
            break;
        }
    }

    case 2:
    case 6:
    case 10:
    case 14:
    {
        if constexpr (sizeof(T) <= 2)
        {
            if (isMLA)
            {
                splitKVCacheForMLAKernel<T, mlaSubWarpSize, 2>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                        tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                        layerNumDomainPP, kvFactor);
            }
            else
            {
                splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 2>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                        tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                        layerNumDomainPP, headNumDomainTP);
            }
            break;
        }
    }
    default:
    {
        if constexpr (sizeof(T) <= 1)
        {
            if (isMLA)
            {
                splitKVCacheForMLAKernel<T, mlaSubWarpSize, 1>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                        tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                        layerNumDomainPP, kvFactor);
            }
            else
            {
                splitKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 1>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputBlockPtrsDev, outputCachePtrsDev,
                        tokensPerBlock, numLayers, headNum, dimsPerHead, inputBlockNum, DomainPPSize, DomainTPSize,
                        layerNumDomainPP, headNumDomainTP);
            }
            break;
        }
        else
        {
            TLLM_THROW("splitKVCacheDispatch encountered an unsupported data type error.");
        }
    }
    }
}

void splitKVCacheDispatch(std::vector<runtime::ITensor::SharedPtr> const& kVCacheBlocks,
    std::vector<runtime::ITensor::SharedPtr>& ouputSplitBlocks, kv_cache::CacheState const& iCacheState,
    kv_cache::CacheState const& oCacheState, int selfIdx, runtime::BufferManager const& bufferManager)
{
    auto dataType = kVCacheBlocks.at(0)->getDataType();
    auto dataSize = tensorrt_llm::common::getDTypeSize(dataType);
    switch (dataSize)
    {
    case 8:
    {
        splitKVCache<int64_t>(kVCacheBlocks, ouputSplitBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 4:
    {
        splitKVCache<int32_t>(kVCacheBlocks, ouputSplitBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 2:
    {
        splitKVCache<int16_t>(kVCacheBlocks, ouputSplitBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 1:
    {
        splitKVCache<int8_t>(kVCacheBlocks, ouputSplitBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    default:
    {
        TLLM_THROW("splitKVCacheDispatch encountered an unsupported data type error.");
    }
    }
}

template <typename T>
void concatKVCache(std::vector<runtime::ITensor::SharedPtr> const& inputSplitBlocks,
    std::vector<runtime::ITensor::SharedPtr>& outputKvCacheBlocks,

    kv_cache::CacheState const& destCacheState, kv_cache::CacheState const& selfCacheState, int selfIdx,
    runtime::BufferManager const& bufferManager)
{

    auto outputBlockNum = outputKvCacheBlocks.size();
    auto targetRankInfo = targetIRanks(destCacheState, selfCacheState, selfIdx);
    TLLM_CHECK(targetRankInfo.mIRanks.size()
        == (static_cast<size_t>(targetRankInfo.mDomainPPSize * targetRankInfo.mDomainTPSize)));

    auto inputCacheNum = targetRankInfo.mIRanks.size();
    if (selfCacheState.getAttentionConfig().mAttentionType == CacheState::AttentionType::kMLA)
    {
        inputCacheNum = targetRankInfo.mDomainPPSize;
    }
    else
    {
        inputCacheNum = inputCacheNum / targetRankInfo.mPeerDupHeadFactor;
    }
    TLLM_CHECK(inputCacheNum == inputSplitBlocks.size());
    TLLM_CHECK(outputBlockNum > 0);
    auto cacheBlockSize = outputKvCacheBlocks.at(0)->getSize();
    auto cacheDataType = outputKvCacheBlocks.at(0)->getDataType();
    std::vector<T*> cachePtrs;
    for (auto&& kvCacheBlock : outputKvCacheBlocks)
    {
        TLLM_CHECK(kvCacheBlock->getDataType() == cacheDataType);
        TLLM_CHECK(kvCacheBlock->getSize() == cacheBlockSize);
        cachePtrs.push_back(static_cast<T*>(kvCacheBlock->data()));
    }
    for (auto&& inputSplitBlock : inputSplitBlocks)
    {
        TLLM_CHECK(inputSplitBlock->getDataType() == cacheDataType);
        TLLM_CHECK(inputSplitBlock->getSize() == cacheBlockSize * outputBlockNum / inputCacheNum);
        cachePtrs.push_back(static_cast<T*>(inputSplitBlock->data()));
    }
    runtime::BufferManager::IBufferPtr PtrsDeviceBuffer
        = bufferManager.gpu(cachePtrs.size(), nvinfer1::DataType::kINT64);
    TLLM_CHECK(PtrsDeviceBuffer->getSizeInBytes() == cachePtrs.size() * sizeof(T*));
    bufferManager.copy(cachePtrs.data(), *PtrsDeviceBuffer, runtime::MemoryType::kCPU);

    constexpr int subWarpSize = 8;
    constexpr int subWarpNumInGroup = 8;
    int blockDimx = 128;

    auto const& selfParallelConfig = selfCacheState.getParallelConfig();
    auto const& selfModelConfig = selfCacheState.getModelConfig();
    auto const& destParallelConfig = destCacheState.getParallelConfig();
    auto const& selfAttentionConfig = selfCacheState.getAttentionConfig();

    int oPPNum = selfParallelConfig.mPipelineParallelism;
    // layers
    unsigned int gridDimx = selfModelConfig.mNbKvHeadsPerLayer.size() / oPPNum;
    // blockNum
    unsigned int gridDimy = outputBlockNum;

    dim3 gridDim{gridDimx, gridDimy};
    int const sizePerHead = selfModelConfig.mSizePerHead;
    int const endLayerId = selfModelConfig.mNbKvHeadsPerLayer.size() / oPPNum;
    T** ouptutBlockPtrsDev = static_cast<T**>(PtrsDeviceBuffer->data());
    T const** inputSplitBlockPtrsDev = static_cast<T const**>(PtrsDeviceBuffer->data()) + outputBlockNum;
    int const tokensPerBlock = selfModelConfig.mTokensPerBlock;
    int const numLayers = selfModelConfig.mNbKvHeadsPerLayer.size() / oPPNum;
    int const headNum = selfModelConfig.mNbKvHeadsPerLayer[0];
    int const dimsPerHead = selfModelConfig.mSizePerHead;
    int const DomainPPSize = targetRankInfo.mDomainPPSize;
    int const DomainTPSize = targetRankInfo.mDomainTPSize;

    int const layerNumDomainPP = numLayers / DomainPPSize;
    int const headNumDomainTP
        = headNum / (DomainTPSize / targetRankInfo.mPeerDupHeadFactor); // TODO: duplicate head factor
    int const kvFactor = selfAttentionConfig.mKvFactor;

    bool isMLA = selfAttentionConfig.mAttentionType == CacheState::AttentionType::kMLA;
    TLLM_LOG_DEBUG(
        "concatKVCache - numLayers: %d, headNum: %d, domainPPSize: %d, domainTPSize: %d, "
        "layersPerDomainPP: %d, headsPerDomainTP: %d",
        numLayers, headNum, DomainPPSize, DomainTPSize, layerNumDomainPP, headNumDomainTP);

    int const remainder = sizePerHead * sizeof(T) % 16;

    int const mlaSubWarpSize = 16;
    switch (remainder)
    {
    case 0:
    {
        if (isMLA)
        {
            concatKVCacheForMLAKernel<T, mlaSubWarpSize, 16>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev, ouptutBlockPtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, kvFactor);
        }
        else
        {
            concatKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 16>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev, ouptutBlockPtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);
        }
        break;
    }
    case 8:
    {
        if (isMLA)
        {
            concatKVCacheForMLAKernel<T, mlaSubWarpSize, 8><<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(
                inputSplitBlockPtrsDev, ouptutBlockPtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead,
                outputBlockNum, DomainPPSize, DomainTPSize, layerNumDomainPP, kvFactor);
        }
        else
        {
            concatKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 8>
                <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev, ouptutBlockPtrsDev,
                    tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum, DomainPPSize, DomainTPSize,
                    layerNumDomainPP, headNumDomainTP);
        }
        break;
    }
    case 4:
    case 12:
    {
        if constexpr (sizeof(T) <= 4)
        {
            if (isMLA)
            {
                concatKVCacheForMLAKernel<T, mlaSubWarpSize, 4>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev,
                        ouptutBlockPtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum,
                        DomainPPSize, DomainTPSize, layerNumDomainPP, kvFactor);
            }
            else
            {
                concatKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 4>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev,
                        ouptutBlockPtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum,
                        DomainPPSize, DomainTPSize, layerNumDomainPP, headNumDomainTP);
            }

            break;
        }
    }
    case 2:
    case 6:
    case 10:
    case 14:
    {
        if constexpr (sizeof(T) <= 2)
        {
            if (isMLA)
            {
                concatKVCacheForMLAKernel<T, mlaSubWarpSize, 2>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev,
                        ouptutBlockPtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum,
                        DomainPPSize, DomainTPSize, layerNumDomainPP, kvFactor);
            }
            else
            {
                concatKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 2>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev,
                        ouptutBlockPtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum,
                        DomainPPSize, DomainTPSize, layerNumDomainPP, headNumDomainTP);
            }
            break;
        }
    }
    default:
    {
        if constexpr (sizeof(T) <= 1)
        {
            if (isMLA)
            {
                concatKVCacheForMLAKernel<T, mlaSubWarpSize, 1>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev,
                        ouptutBlockPtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum,
                        DomainPPSize, DomainTPSize, layerNumDomainPP, kvFactor);
            }
            else
            {
                concatKVCacheKernel<T, subWarpSize, subWarpNumInGroup, 1>
                    <<<gridDim, blockDimx, 0, bufferManager.getStream().get()>>>(inputSplitBlockPtrsDev,
                        ouptutBlockPtrsDev, tokensPerBlock, numLayers, headNum, dimsPerHead, outputBlockNum,
                        DomainPPSize, DomainTPSize, layerNumDomainPP, headNumDomainTP);
            }
            break;
        }
        else
        {
            TLLM_THROW("concatKVCache encountered an unsupported data type error.");
        }
    }
    }
}

void concatKvCacheV2Dispatch(std::vector<runtime::ITensor::SharedPtr> const& inputSplitBlocks,
    std::vector<runtime::ITensor::SharedPtr>& outputKvCacheBlocks, kv_cache::CacheState const& iCacheState,
    kv_cache::CacheState const& oCacheState, int selfIdx, runtime::BufferManager const& bufferManager)
{

    auto dataType = outputKvCacheBlocks.at(0)->getDataType();
    auto dataSize = tensorrt_llm::common::getDTypeSize(dataType);
    switch (dataSize)
    {
    case 8:
    {
        concatKVCache<int64_t>(inputSplitBlocks, outputKvCacheBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 4:
    {
        concatKVCache<int32_t>(inputSplitBlocks, outputKvCacheBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 2:
    {
        concatKVCache<int16_t>(inputSplitBlocks, outputKvCacheBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    case 1:
    {
        concatKVCache<int8_t>(inputSplitBlocks, outputKvCacheBlocks, iCacheState, oCacheState, selfIdx, bufferManager);
        break;
    }
    default:
    {
        TLLM_THROW("concatKVCache encountered an unsupported data type error.");
    }
    }
}

} // namespace tensorrt_llm::executor::kv_cache
