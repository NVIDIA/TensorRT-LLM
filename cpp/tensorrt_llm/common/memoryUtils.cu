#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/assert.h"
#include "tensorrt_llm/common/cudaTypeUtils.cuh"
#include "tensorrt_llm/common/logger.h"
#include "tensorrt_llm/common/memoryUtils.h"

#include <hiprand/hiprand_kernel.h>
#include <sys/stat.h>
#include <unordered_map>

#include <sanitizer/asan_interface.h>

namespace tensorrt_llm
{
namespace common
{

#ifdef __has_feature
#if __has_feature(address_sanitizer)
#define TLLM_HAS_ASAN
#endif
#elif defined(__SANITIZE_ADDRESS__)
#define TLLM_HAS_ASAN
#endif

hipError_t cudaMemcpyAsyncSanitized(
    void* dst, void const* src, size_t count, enum hipMemcpyKind kind, hipStream_t stream)
{
#if defined(TLLM_HAS_ASAN)
    bool needASAN = false;
    if (kind == hipMemcpyDeviceToHost)
    {
        needASAN = true;
    }
    else if (kind == hipMemcpyDefault)
    {
        auto const srcType = getPtrCudaMemoryType(src);
        auto const dstType = getPtrCudaMemoryType(dst);
        needASAN = srcType == hipMemoryTypeDevice && dstType != hipMemoryTypeDevice;
    }

    // Poison the memory area during async copy
    if (needASAN)
    {
        ASAN_POISON_MEMORY_REGION(dst, count);
    }

    auto const result = hipMemcpyAsync(dst, src, count, kind, stream);

    if (result == hipSuccess && needASAN)
    {
        struct ctxType
        {
            void* ptr;
            size_t count;
        };

        auto const ctx = new ctxType{dst, count};
        auto cb = [](hipStream_t, hipError_t, void* data)
        {
            auto const ctx = static_cast<ctxType*>(data);
            ASAN_UNPOISON_MEMORY_REGION(ctx->ptr, ctx->count);
            delete ctx;
        };
        TLLM_CUDA_CHECK(hipStreamAddCallback(stream, cb, ctx, 0));
    }

    return result;
#else
    return hipMemcpyAsync(dst, src, count, kind, stream);
#endif
}

template <typename T>
void deviceMalloc(T** ptr, size_t size, bool is_random_initialize)
{
    check_cuda_error(hipMalloc((void**) (ptr), sizeof(T) * size));
    if (is_random_initialize)
    {
        cudaRandomUniform(*ptr, size);
    }
}

template void deviceMalloc(float** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(half** ptr, size_t size, bool is_random_initialize);
#ifdef ENABLE_BF16
template void deviceMalloc(__hip_bfloat16** ptr, size_t size, bool is_random_initialize);
#endif
template void deviceMalloc(uint16_t** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(bool** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(char** ptr, size_t size, bool is_random_initialize);
template void deviceMalloc(int8_t** ptr, size_t size, bool is_random_initialize);
#ifdef ENABLE_FP8
template void deviceMalloc(__hip_fp8_e4m3_fnuz** ptr, size_t size, bool is_random_initialize);
#endif

template <typename T>
void deviceMemSetZero(T* ptr, size_t size)
{
    check_cuda_error(hipMemset(static_cast<void*>(ptr), 0, sizeof(T) * size));
}

template void deviceMemSetZero(float* ptr, size_t size);
template void deviceMemSetZero(half* ptr, size_t size);
template void deviceMemSetZero(int* ptr, size_t size);
template void deviceMemSetZero(uint32_t* ptr, size_t size);
template void deviceMemSetZero(bool* ptr, size_t size);
#ifdef ENABLE_FP8
template void deviceMemSetZero(__hip_fp8_e4m3_fnuz* ptr, size_t size);
#endif
#ifdef ENABLE_BF16
template void deviceMemSetZero(__hip_bfloat16* ptr, size_t size);
#endif

template <typename T>
void deviceFree(T*& ptr)
{
    if (ptr != NULL)
    {
        check_cuda_error(hipFree(ptr));
        ptr = NULL;
    }
}

template void deviceFree(float*& ptr);
template void deviceFree(half*& ptr);
#ifdef ENABLE_BF16
template void deviceFree(__hip_bfloat16*& ptr);
#endif
template void deviceFree(unsigned short*& ptr);
template void deviceFree(int*& ptr);
template void deviceFree(bool*& ptr);
template void deviceFree(char*& ptr);
template void deviceFree(int8_t*& ptr);
#ifdef ENABLE_FP8
template void deviceFree(__hip_fp8_e4m3_fnuz*& ptr);
#endif

template <typename T>
void deviceFill(T* devptr, size_t size, T value, hipStream_t stream)
{
    T* arr = new T[size];
    std::fill(arr, arr + size, value);
    check_cuda_error(hipMemcpyAsync(devptr, arr, sizeof(T) * size, hipMemcpyHostToDevice, stream));
    delete[] arr;
}

template void deviceFill(float* devptr, size_t size, float value, hipStream_t stream);
template void deviceFill(half* devptr, size_t size, half value, hipStream_t stream);
#ifdef ENABLE_BF16
template void deviceFill(__hip_bfloat16* devptr, size_t size, __hip_bfloat16 value, hipStream_t stream);
#endif
template void deviceFill(int* devptr, size_t size, int value, hipStream_t stream);
template void deviceFill(bool* devptr, size_t size, bool value, hipStream_t stream);

template <typename T>
void cudaD2Hcpy(T* tgt, T const* src, const size_t size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDeviceToHost));
}

template void cudaD2Hcpy(float* tgt, float const* src, size_t size);
template void cudaD2Hcpy(half* tgt, half const* src, size_t size);
#ifdef ENABLE_BF16
template void cudaD2Hcpy(__hip_bfloat16* tgt, __hip_bfloat16 const* src, size_t size);
#endif
template void cudaD2Hcpy(int* tgt, int const* src, size_t size);
template void cudaD2Hcpy(bool* tgt, bool const* src, size_t size);
#ifdef ENABLE_FP8
template void cudaD2Hcpy(__hip_fp8_e4m3_fnuz* tgt, __hip_fp8_e4m3_fnuz const* src, size_t size);
#endif
template void cudaD2Hcpy(unsigned long long* tgt, unsigned long long const* src, size_t size);
template void cudaD2Hcpy(unsigned int* tgt, unsigned int const* src, size_t size);
template void cudaD2Hcpy(int8_t* tgt, int8_t const* src, size_t size);

template <typename T>
void cudaH2Dcpy(T* tgt, T const* src, const size_t size)
{
    check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyHostToDevice));
}

template void cudaH2Dcpy(float* tgt, float const* src, size_t size);
template void cudaH2Dcpy(half* tgt, half const* src, size_t size);
#ifdef ENABLE_BF16
template void cudaH2Dcpy(__hip_bfloat16* tgt, __hip_bfloat16 const* src, size_t size);
#endif
template void cudaH2Dcpy(int* tgt, int const* src, size_t size);
template void cudaH2Dcpy(bool* tgt, bool const* src, size_t size);
#ifdef ENABLE_FP8
template void cudaH2Dcpy(__hip_fp8_e4m3_fnuz* tgt, __hip_fp8_e4m3_fnuz const* src, size_t size);
#endif
template void cudaH2Dcpy(unsigned long long* tgt, unsigned long long const* src, size_t size);
template void cudaH2Dcpy(unsigned int* tgt, unsigned int const* src, size_t size);
template void cudaH2Dcpy(int8_t* tgt, int8_t const* src, size_t size);

template <typename T>
void cudaD2Dcpy(T* tgt, T const* src, const size_t size, hipStream_t stream)
{
    check_cuda_error(hipMemcpyAsync(tgt, src, sizeof(T) * size, hipMemcpyDeviceToDevice, stream));
}

template void cudaD2Dcpy(float* tgt, float const* src, size_t size, hipStream_t stream);
template void cudaD2Dcpy(half* tgt, half const* src, size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaD2Dcpy(__hip_bfloat16* tgt, __hip_bfloat16 const* src, size_t size, hipStream_t stream);
#endif
template void cudaD2Dcpy(int* tgt, int const* src, size_t size, hipStream_t stream);
template void cudaD2Dcpy(bool* tgt, bool const* src, size_t size, hipStream_t stream);
template void cudaD2Dcpy(int8_t* tgt, int8_t const* src, size_t size, hipStream_t stream);
#ifdef ENABLE_FP8
template void cudaD2Dcpy(__hip_fp8_e4m3_fnuz* tgt, __hip_fp8_e4m3_fnuz const* src, size_t size, hipStream_t stream);
#endif
template void cudaD2Dcpy(unsigned long long* tgt, unsigned long long const* src, size_t size, hipStream_t stream);

template <typename T_OUT, typename T_IN>
__global__ void cudaCast(T_OUT* dst, T_IN* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x)
    {
        dst[tid] = (T_OUT) ((float) (src[tid]));
    }
}

template <typename T_OUT, typename T_IN>
void invokeCudaCast(T_OUT* dst, T_IN const* const src, const size_t size, hipStream_t stream)
{
    cudaCast<<<256, 256, 0, stream>>>(dst, src, size);
}

template void invokeCudaCast(float* dst, half const* const src, const size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeCudaCast(float* dst, __hip_bfloat16 const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(__hip_bfloat16* dst, float const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(__hip_bfloat16* dst, half const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(half* dst, __hip_bfloat16 const* const src, const size_t size, hipStream_t stream);
#endif
#ifdef ENABLE_FP8
template void invokeCudaCast(float* dst, __hip_fp8_e4m3_fnuz const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(
    __hip_bfloat16* dst, __hip_fp8_e4m3_fnuz const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(half* dst, __hip_fp8_e4m3_fnuz const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(__hip_fp8_e4m3_fnuz* dst, float const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(
    __hip_fp8_e4m3_fnuz* dst, __hip_bfloat16 const* const src, const size_t size, hipStream_t stream);
template void invokeCudaCast(__hip_fp8_e4m3_fnuz* dst, half const* const src, const size_t size, hipStream_t stream);
#endif

template <typename T>
void cudaAutoCpy(T* tgt, T const* src, const size_t size, hipStream_t stream)
{
    if (stream != NULL)
    {
        check_cuda_error(cudaMemcpyAsyncSanitized(tgt, src, sizeof(T) * size, hipMemcpyDefault, stream));
    }
    else
    {
        check_cuda_error(hipMemcpy(tgt, src, sizeof(T) * size, hipMemcpyDefault));
    }
}

template void cudaAutoCpy(float* tgt, float const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(half* tgt, half const* src, size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaAutoCpy(__hip_bfloat16* tgt, __hip_bfloat16 const* src, size_t size, hipStream_t stream);
#endif
template void cudaAutoCpy(int* tgt, int const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(bool* tgt, bool const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(int8_t* tgt, int8_t const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(uint8_t* tgt, uint8_t const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(uint32_t* tgt, uint32_t const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(unsigned long long* tgt, unsigned long long const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(unsigned long* tgt, unsigned long const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(char* tgt, char const* src, size_t size, hipStream_t stream);

template void cudaAutoCpy(float const** tgt, float const* const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(half const** tgt, half const* const* src, size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void cudaAutoCpy(__hip_bfloat16 const** tgt, __hip_bfloat16 const* const* src, size_t size, hipStream_t stream);
#endif
template void cudaAutoCpy(int const** tgt, int const* const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(bool const** tgt, bool const* const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(int8_t const** tgt, int8_t const* const* src, size_t size, hipStream_t stream);
template void cudaAutoCpy(
    unsigned long long const** tgt, unsigned long long const* const* src, size_t size, hipStream_t stream);

template <typename T>
__global__ void cuda_random_uniform_kernel(T* buffer, const size_t size, int const seq_offset)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((unsigned long long int) 1337, idx + seq_offset, 0, &local_state);
    for (size_t index = idx; index < size; index += blockDim.x * gridDim.x)
    {
        buffer[index] = (T) (hiprand_uniform(&local_state) * 0.2f - 0.1f);
    }
}

template <>
__global__ void cuda_random_uniform_kernel<int>(int* buffer, const size_t size, int const seq_offset)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((float) 1337.f, idx + seq_offset, 0, &local_state);
    for (size_t index = idx; index < size; index += blockDim.x * gridDim.x)
    {
        buffer[index] = hiprand(&local_state);
    }
}

template <>
__global__ void cuda_random_uniform_kernel<bool>(bool* buffer, const size_t size, int const seq_offset)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((float) 1337.f, idx + seq_offset, 0, &local_state);
    for (size_t index = idx; index < size; index += blockDim.x * gridDim.x)
    {
        buffer[index] = (hiprand(&local_state) % 2 == 0);
    }
}

template <>
__global__ void cuda_random_uniform_kernel<char>(char* buffer, const size_t size, int const seq_offset)
{
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprandState_t local_state;
    hiprand_init((float) 1337.f, idx + seq_offset, 0, &local_state);
    for (size_t index = idx; index < size; index += blockDim.x * gridDim.x)
    {
        buffer[index] = hiprand(&local_state) % 0xFF;
    }
}

template <typename T>
void cudaRandomUniform(T* buffer, const size_t size)
{
    static int seq_offset = 0;
    cuda_random_uniform_kernel<T><<<256, 256>>>(buffer, size, seq_offset);
    seq_offset += 256 * 256;
}

template void cudaRandomUniform(float* buffer, const size_t size);
template void cudaRandomUniform(half* buffer, const size_t size);
#ifdef ENABLE_BF16
template void cudaRandomUniform(__hip_bfloat16* buffer, const size_t size);
#endif
template void cudaRandomUniform(int* buffer, const size_t size);
template void cudaRandomUniform(bool* buffer, const size_t size);
template void cudaRandomUniform(char* buffer, const size_t size);
#ifdef ENABLE_FP8
template void cudaRandomUniform(__hip_fp8_e4m3_fnuz* buffer, const size_t size);
#endif

// loads data from binary file. If it succeeds, returns a non-empty vector. If loading fails or
// the product of the elements in shape is 0, this function will return an empty vector.
template <typename T>
std::vector<T> loadWeightFromBinHelper(std::vector<size_t> shape, std::string filename)
{
    if (shape.size() > 2)
    {
        printf("[ERROR] shape should have less than two dims \n");
        return std::vector<T>();
    }
    size_t dim0 = shape[0], dim1 = 1;
    if (shape.size() == 2)
    {
        dim1 = shape[1];
    }
    size_t size = dim0 * dim1;
    if (size == 0)
    {
        TLLM_LOG_WARNING("shape is zero, skip loading weight from file %s \n", filename.c_str());
        return std::vector<T>();
    }

    std::vector<T> host_array(size);
    std::ifstream in(filename, std::ios::in | std::ios::binary);
    if (!in.is_open())
    {
        TLLM_LOG_WARNING("file %s cannot be opened, loading model fails! \n", filename.c_str());
        return std::vector<T>();
    }

    size_t loaded_data_size = sizeof(T) * size;
    in.seekg(0, in.end);
    in.seekg(0, in.beg);

    TLLM_LOG_DEBUG("Read " + std::to_string(loaded_data_size) + " bytes from " + filename);
    in.read((char*) host_array.data(), loaded_data_size);

    size_t in_get_size = in.gcount();
    if (in_get_size != loaded_data_size)
    {
        TLLM_LOG_WARNING("file %s only has %ld, but request %ld, loading model fails! \n", filename.c_str(),
            in_get_size, loaded_data_size);
        return std::vector<T>();
    }
    in.close();
    // If we succeed, return an array with values.
    return host_array;
}

template <typename T, typename T_IN>
int loadWeightFromBinFunc(T* ptr, std::vector<size_t> shape, std::string filename)
{
    std::vector<T_IN> host_array = loadWeightFromBinHelper<T_IN>(shape, filename);

    if (host_array.empty())
    {
        return 0;
    }

    if (std::is_same<T, T_IN>::value == true)
    {
        cudaH2Dcpy(ptr, (T*) host_array.data(), host_array.size());
    }
    else
    {
        T_IN* ptr_2 = nullptr;
        deviceMalloc(&ptr_2, host_array.size(), false);
        cudaH2Dcpy(ptr_2, host_array.data(), host_array.size());
        invokeCudaD2DcpyConvert(ptr, ptr_2, host_array.size());
        deviceFree(ptr_2);
    }
    return 0;
}

template int loadWeightFromBinFunc<float, float>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, float>(half* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<float, half>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, half>(half* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<int8_t, int8_t>(int8_t* ptr, std::vector<size_t> shape, std::string filename);
#ifdef ENABLE_BF16
template int loadWeightFromBinFunc<__hip_bfloat16, float>(
    __hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<__hip_bfloat16, half>(
    __hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<float, __hip_bfloat16>(float* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<half, __hip_bfloat16>(half* ptr, std::vector<size_t> shape, std::string filename);
template int loadWeightFromBinFunc<__hip_bfloat16, __hip_bfloat16>(
    __hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename);
#endif // ENABLE_BF16
template int loadWeightFromBinFunc<int, int>(int* ptr, std::vector<size_t> shape, std::string filename);
#ifdef ENABLE_FP8
template int loadWeightFromBinFunc<__hip_fp8_e4m3_fnuz, float>(
    __hip_fp8_e4m3_fnuz* ptr, std::vector<size_t> shape, std::string filename);
#endif // ENABLE_FP8

template <typename T>
int loadWeightFromBin(T* ptr, std::vector<size_t> shape, std::string filename, TRTLLMCudaDataType model_file_type)
{
    switch (model_file_type)
    {
    case TRTLLMCudaDataType::FP32: loadWeightFromBinFunc<T, float>(ptr, shape, filename); break;
    case TRTLLMCudaDataType::FP16: loadWeightFromBinFunc<T, half>(ptr, shape, filename); break;
    case TRTLLMCudaDataType::INT8: loadWeightFromBinFunc<T, int8_t>(ptr, shape, filename); break;
#ifdef ENABLE_BF16
    case TRTLLMCudaDataType::BF16: loadWeightFromBinFunc<T, __hip_bfloat16>(ptr, shape, filename); break;
#endif
#ifdef ENABLE_FP8
    case TRTLLMCudaDataType::FP8: loadWeightFromBinFunc<T, float>(ptr, shape, filename); break;
#endif
    default: TLLM_LOG_ERROR("Does not support TRTLLMCudaDataType=%d", model_file_type); TLLM_CHECK(false);
    }
    return 0;
}

template <>
int loadWeightFromBin(int* ptr, std::vector<size_t> shape, std::string filename, TRTLLMCudaDataType model_file_type)
{
    loadWeightFromBinFunc<int, int>(ptr, shape, filename);
    return 0;
}

template int loadWeightFromBin(
    float* ptr, std::vector<size_t> shape, std::string filename, TRTLLMCudaDataType model_file_type);
template int loadWeightFromBin(
    half* ptr, std::vector<size_t> shape, std::string filename, TRTLLMCudaDataType model_file_type);
template int loadWeightFromBin(
    int8_t* ptr, std::vector<size_t> shape, std::string filename, TRTLLMCudaDataType model_file_type);
#ifdef ENABLE_BF16
template int loadWeightFromBin(
    __hip_bfloat16* ptr, std::vector<size_t> shape, std::string filename, TRTLLMCudaDataType model_file_type);
#endif
#ifdef ENABLE_FP8
template int loadWeightFromBin(
    __hip_fp8_e4m3_fnuz* ptr, std::vector<size_t> shape, std::string filename, TRTLLMCudaDataType model_file_type);
#endif

template <typename T_IN, typename T_OUT>
__global__ void cudaD2DcpyConvert(T_OUT* dst, const T_IN* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x)
    {
        dst[tid] = cuda_cast<T_OUT>(src[tid]);
    }
}

template <typename T_IN, typename T_OUT>
void invokeCudaD2DcpyConvert(T_OUT* tgt, const T_IN* src, const size_t size, hipStream_t stream)
{
    cudaD2DcpyConvert<<<256, 256, 0, stream>>>(tgt, src, size);
}

template void invokeCudaD2DcpyConvert(int8_t* tgt, float const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, int8_t const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, int const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(half* tgt, int const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, float const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(half* tgt, float const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, half const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(uint32_t* tgt, int const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(int* tgt, uint32_t const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(int* tgt, float const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(int* tgt, half const* src, const size_t size, hipStream_t stream);

#ifdef ENABLE_BF16
template void invokeCudaD2DcpyConvert(__hip_bfloat16* tgt, float const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(__hip_bfloat16* tgt, int const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(float* tgt, __hip_bfloat16 const* src, const size_t size, hipStream_t stream);
template void invokeCudaD2DcpyConvert(int* tgt, __hip_bfloat16 const* src, const size_t size, hipStream_t stream);
#endif // ENABLE_BF16

template <typename T_IN, typename T_OUT>
__global__ void cudaD2DScaleCpyConvert(
    T_OUT* dst, const T_IN* src, float const* scale, bool invert_scale, const size_t size)
{
    float const scale_value = invert_scale ? 1.0f / scale[0] : scale[0];
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x)
    {
        dst[tid] = cuda_cast<T_OUT>(cuda_cast<float>(src[tid]) * scale_value);
    }
}

template <typename T_IN, typename T_OUT>
void invokeCudaD2DScaleCpyConvert(
    T_OUT* tgt, const T_IN* src, float const* scale, bool invert_scale, const size_t size, hipStream_t stream)
{
    cudaD2DScaleCpyConvert<<<256, 256, 0, stream>>>(tgt, src, scale, invert_scale, size);
}

// clang-format off
template void invokeCudaD2DScaleCpyConvert(float* tgt, const int32_t* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
template void invokeCudaD2DScaleCpyConvert(int32_t* tgt, const float* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
template void invokeCudaD2DScaleCpyConvert(half* tgt, const int32_t* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
template void invokeCudaD2DScaleCpyConvert(int32_t* tgt, const half* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeCudaD2DScaleCpyConvert(__hip_bfloat16* tgt, const int32_t* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
template void invokeCudaD2DScaleCpyConvert(int32_t* tgt, const __hip_bfloat16* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
#endif  // ENABLE_BF16
#ifdef ENABLE_FP8
template void invokeCudaD2DScaleCpyConvert(float* tgt, const __hip_fp8_e4m3_fnuz* src, const float* scale, bool invert_scale, const size_t size, hipStream_t stream);
#endif  // ENABLE_FP8
// clang-format on

void invokeCudaD2DcpyHalf2Float(float* dst, half* src, const size_t size, hipStream_t stream)
{
    invokeCudaD2DcpyConvert(dst, src, size, stream);
}

void invokeCudaD2DcpyFloat2Half(half* dst, float* src, const size_t size, hipStream_t stream)
{
    invokeCudaD2DcpyConvert(dst, src, size, stream);
}

template <typename T>
void saveToBinary(T const* ptr, const size_t size, std::string filename)
{

    std::vector<T> h_ptr(size);
    cudaD2Hcpy(h_ptr.data(), ptr, size);
    std::vector<float> float_ptr(size);
    for (size_t i = 0; i < size; i++)
    {
        float_ptr[i] = (float) h_ptr[i];
    }

    std::ofstream out(filename, std::ios::out | std::ios::binary);
    TLLM_CHECK_WITH_INFO(out.is_open(), "Fail to open file " + filename);

    out.write((char*) float_ptr.data(), size * sizeof(float));
}

template void saveToBinary(float const* ptr, const size_t size, std::string filename);
template void saveToBinary(half const* ptr, const size_t size, std::string filename);
#ifdef ENABLE_BF16
template void saveToBinary(__hip_bfloat16 const* ptr, const size_t size, std::string filename);
#endif // ENABLE_BF16

template <>
void saveToBinary(int const* ptr, const size_t size, std::string filename)
{
    std::vector<int> h_ptr(size);
    cudaD2Hcpy(h_ptr.data(), ptr, size);
    std::ofstream out(filename, std::ios::out | std::ios::binary);
    TLLM_CHECK_WITH_INFO(out.is_open(), "Fail to open file " + filename);
    out.write((char*) h_ptr.data(), size * sizeof(int));
}

template <typename T_IN, typename T_fake_type>
__global__ void fakeCast(T_IN* input_ptr, const size_t size)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
    {
        T_fake_type tmp_val = (T_fake_type) ((float) input_ptr[i]);
        input_ptr[i] = (T_IN) ((float) tmp_val);
    }
}

template <typename T_IN, typename T_fake_type>
void invokeFakeCast(T_IN* input_ptr, const size_t size, hipStream_t stream)
{
    dim3 block(256);
    dim3 grid((size + 255) / 256);
    fakeCast<T_IN, T_fake_type><<<grid, block, 0, stream>>>(input_ptr, size);
}

#ifdef ENABLE_FP8
__global__ void cudaD2Dcpyfp82Float(float* dst, __hip_fp8_e4m3_fnuz* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x)
    {
        dst[tid] = (float) (src[tid]);
    }
}

void invokeCudaD2Dcpyfp82Float(float* dst, __hip_fp8_e4m3_fnuz* src, const size_t size, hipStream_t stream)
{
    cudaD2Dcpyfp82Float<<<256, 256, 0, stream>>>(dst, src, size);
}

__global__ void cudaD2Dcpyfp82Half(half* dst, __hip_fp8_e4m3_fnuz* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x)
    {
        dst[tid] = (half) ((float) (src[tid]));
    }
}

void invokeCudaD2Dcpyfp82Half(half* dst, __hip_fp8_e4m3_fnuz* src, const size_t size, hipStream_t stream)
{
    cudaD2Dcpyfp82Half<<<256, 256, 0, stream>>>(dst, src, size);
}

__global__ void cudaD2DcpyFloat2fp8(__hip_fp8_e4m3_fnuz* dst, float* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x)
    {
        dst[tid] = (__hip_fp8_e4m3_fnuz) src[tid];
    }
}

void invokeCudaD2DcpyFloat2fp8(__hip_fp8_e4m3_fnuz* dst, float* src, const size_t size, hipStream_t stream)
{
    cudaD2DcpyFloat2fp8<<<256, 256, 0, stream>>>(dst, src, size);
}

__global__ void cudaD2DcpyHalf2fp8(__hip_fp8_e4m3_fnuz* dst, half* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x)
    {
        dst[tid] = (__hip_fp8_e4m3_fnuz) src[tid];
    }
}

void invokeCudaD2DcpyHalf2fp8(__hip_fp8_e4m3_fnuz* dst, half* src, const size_t size, hipStream_t stream)
{
    cudaD2DcpyHalf2fp8<<<256, 256, 0, stream>>>(dst, src, size);
}

__global__ void cudaD2DcpyBfloat2fp8(__hip_fp8_e4m3_fnuz* dst, __hip_bfloat16* src, const size_t size)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < size; tid += blockDim.x * gridDim.x)
    {
        dst[tid] = (__hip_fp8_e4m3_fnuz) src[tid];
    }
}

void invokeCudaD2DcpyBfloat2fp8(__hip_fp8_e4m3_fnuz* dst, __hip_bfloat16* src, const size_t size, hipStream_t stream)
{
    cudaD2DcpyBfloat2fp8<<<256, 256, 0, stream>>>(dst, src, size);
}

#endif // ENABLE_FP8

template <typename T_OUT, typename T_IN>
__global__ void transpose(T_OUT* dst, T_IN* src, const size_t dim0, const size_t dim1)
{
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < dim0 * dim1; tid += blockDim.x * gridDim.x)
    {
        const size_t src_col_id = tid % dim1;
        const size_t src_row_id = tid / dim1;
        dst[src_col_id * dim0 + src_row_id] = (T_OUT) (src[tid]);
    }
}

template <typename T>
void invokeInPlaceTranspose(T* data, T* workspace, const size_t dim0, const size_t dim1)
{
    // copy data to workspace, and then transpose from workspace to data
    cudaD2Dcpy(workspace, data, dim0 * dim1);
    transpose<<<256, 256>>>(data, workspace, dim0, dim1);
}

#ifdef ENABLE_FP8
template void invokeInPlaceTranspose(
    __hip_fp8_e4m3_fnuz* data, __hip_fp8_e4m3_fnuz* workspace, const size_t dim0, const size_t dim1);
#endif // ENABLE_FP8
#ifdef ENABLE_BF16
template void invokeInPlaceTranspose(
    __hip_bfloat16* data, __hip_bfloat16* workspace, const size_t dim0, const size_t dim1);
#endif // ENABLE_BF16
template void invokeInPlaceTranspose(float* data, float* workspace, const size_t dim0, const size_t dim1);

template <typename T_OUT, typename T_IN>
__global__ void transpose0213(
    T_OUT* dst, T_IN* src, const size_t dim0, const size_t dim1, const size_t dim2, const size_t dim3)
{
    // src permutation: [0, 1, 2, 3]
    // dst permutation: [0, 2, 1, 3]
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < dim0 * dim1 * dim2 * dim3;
         tid += blockDim.x * gridDim.x)
    {
        size_t tmp_idx = tid;
        const size_t dim_3_idx = tmp_idx % dim3;
        tmp_idx = (tmp_idx - dim_3_idx) / dim3;
        const size_t dim_2_idx = tmp_idx % dim2;
        tmp_idx = (tmp_idx - dim_2_idx) / dim2;
        const size_t dim_1_idx = tmp_idx % dim1;
        tmp_idx = (tmp_idx - dim_1_idx) / dim1;
        const size_t dim_0_idx = tmp_idx % dim0;
        dst[dim_0_idx * dim1 * dim2 * dim3 + dim_2_idx * dim1 * dim3 + dim_1_idx * dim3 + dim_3_idx] = src[tid];
    }
}

template <typename T>
void invokeInPlaceTranspose0213(
    T* data, T* workspace, const size_t dim0, const size_t dim1, const size_t dim2, const size_t dim3)
{
    // copy data to workspace, and then transpose from workspace to data
    // Note that this kernel is used for pre-processing and not very efficient.
    cudaD2Dcpy(workspace, data, dim0 * dim1 * dim2 * dim3);
    transpose0213<<<256, 256>>>(data, workspace, dim0, dim1, dim2, dim3);
}

#ifdef ENABLE_FP8
template void invokeInPlaceTranspose0213(__hip_fp8_e4m3_fnuz* data, __hip_fp8_e4m3_fnuz* workspace, const size_t dim0,
    const size_t dim1, const size_t dim2, const size_t dim3);
#endif // ENABLE_FP8
#ifdef ENABLE_BF16
template void invokeInPlaceTranspose0213(__hip_bfloat16* data, __hip_bfloat16* workspace, const size_t dim0,
    const size_t dim1, const size_t dim2, const size_t dim3);
#endif // ENABLE_BF16
template void invokeInPlaceTranspose0213(
    float* data, float* workspace, const size_t dim0, const size_t dim1, const size_t dim2, const size_t dim3);

template <typename T_OUT, typename T_IN>
__global__ void transpose102(T_OUT* dst, T_IN* src, const size_t dim0, const size_t dim1, const size_t dim2)
{
    // src permutation: [0, 1, 2]
    // dst permutation: [1, 0, 2]
    for (size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < dim0 * dim1 * dim2; tid += blockDim.x * gridDim.x)
    {
        size_t tmp_idx = tid;
        const size_t dim_2_idx = tmp_idx % dim2;
        tmp_idx = (tmp_idx - dim_2_idx) / dim2;
        const size_t dim_1_idx = tmp_idx % dim1;
        tmp_idx = (tmp_idx - dim_1_idx) / dim1;
        const size_t dim_0_idx = tmp_idx % dim0;
        dst[dim_1_idx * dim0 * dim2 + dim_0_idx * dim2 + dim_2_idx] = src[tid];
    }
}

template <typename T>
void invokeInPlaceTranspose102(T* data, T* workspace, const size_t dim0, const size_t dim1, const size_t dim2)
{
    // copy data to workspace, and then transpose from workspace to data
    // Note that this kernel is used for pre-processing and not very efficient.
    cudaD2Dcpy(workspace, data, dim0 * dim1 * dim2);
    transpose102<<<256, 256>>>(data, workspace, dim0, dim1, dim2);
}

#ifdef ENABLE_FP8
template void invokeInPlaceTranspose102(
    __hip_fp8_e4m3_fnuz* data, __hip_fp8_e4m3_fnuz* workspace, const size_t dim0, const size_t dim1, const size_t dim2);
#endif // ENABLE_FP8
#ifdef ENABLE_BF16
template void invokeInPlaceTranspose102(
    __hip_bfloat16* data, __hip_bfloat16* workspace, const size_t dim0, const size_t dim1, const size_t dim2);
#endif // ENABLE_BF16
template void invokeInPlaceTranspose102(
    float* data, float* workspace, const size_t dim0, const size_t dim1, const size_t dim2);

template <typename T>
void __global__ multiplyScale(T* tensor, float scale, const size_t size)
{
    for (size_t index = threadIdx.x + blockIdx.x * blockDim.x; index < size; index += blockDim.x * gridDim.x)
    {
        tensor[index] = (T) (((float) tensor[index]) * scale);
    }
}

template <typename T>
void invokeMultiplyScale(T* tensor, float scale, const size_t size, hipStream_t stream)
{
    int block = 256;
    int grid = (size + 255) / 256;
    multiplyScale<<<grid, block, 0, stream>>>(tensor, scale, size);
}

template void invokeMultiplyScale(float* tensor, float scale, const size_t size, hipStream_t stream);
template void invokeMultiplyScale(half* tensor, float scale, const size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeMultiplyScale(__hip_bfloat16* tensor, float scale, const size_t size, hipStream_t stream);
#endif
#ifdef ENABLE_FP8
template void invokeMultiplyScale(__hip_fp8_e4m3_fnuz* tensor, float scale, const size_t size, hipStream_t stream);
#endif

template <typename T>
void __global__ divideScale(T* tensor, float scale, const size_t size)
{
    for (size_t index = threadIdx.x + blockIdx.x * blockDim.x; index < size; index += blockDim.x * gridDim.x)
    {
        tensor[index] = (T) (((float) tensor[index]) / scale);
    }
}

template <typename T>
void invokeDivideScale(T* tensor, float scale, const size_t size, hipStream_t stream)
{
    int block = 256;
    int grid = (size + 255) / 256;
    divideScale<<<grid, block, 0, stream>>>(tensor, scale, size);
}

template void invokeDivideScale(float* tensor, float scale, const size_t size, hipStream_t stream);
template void invokeDivideScale(half* tensor, float scale, const size_t size, hipStream_t stream);
#ifdef ENABLE_BF16
template void invokeDivideScale(__hip_bfloat16* tensor, float scale, const size_t size, hipStream_t stream);
#endif
#ifdef ENABLE_FP8
template void invokeDivideScale(__hip_fp8_e4m3_fnuz* tensor, float scale, const size_t size, hipStream_t stream);
#endif
#ifdef ENABLE_BF16
template void invokeFakeCast<float, __hip_bfloat16>(float* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<__hip_bfloat16, __hip_bfloat16>(
    __hip_bfloat16* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<half, __hip_bfloat16>(half* input_ptr, const size_t size, hipStream_t stream);
#endif
template void invokeFakeCast<float, half>(float* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<float, float>(float* input_ptr, const size_t size, hipStream_t stream);
#ifdef ENABLE_FP8
template void invokeFakeCast<float, __hip_fp8_e4m3_fnuz>(float* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<half, __hip_fp8_e4m3_fnuz>(half* input_ptr, const size_t size, hipStream_t stream);
template void invokeFakeCast<__hip_bfloat16, __hip_fp8_e4m3_fnuz>(
    __hip_bfloat16* input_ptr, const size_t size, hipStream_t stream);
#endif

size_t cuda_datatype_size(TRTLLMCudaDataType dt)
{
    static const std::unordered_map<TRTLLMCudaDataType, size_t> sizes{
        {TRTLLMCudaDataType::FP32, sizeof(float)}, {TRTLLMCudaDataType::FP16, sizeof(half)}
#ifdef ENABLE_BF16
        ,
        {TRTLLMCudaDataType::BF16, sizeof(__hip_bfloat16)}
#endif
    };

    return sizes.at(dt);
}

template <typename T>
__global__ void check_range(T const* buffer, size_t size, T min, T max, bool* d_within_range)
{
    for (size_t i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
    {
        const T val = buffer[i];
        if (val < min || val > max)
        {
            *d_within_range = false;
        }
    }
}

template <typename T>
bool invokeCheckRange(T const* buffer, const size_t size, T min, T max, bool* d_within_range, hipStream_t stream)
{
    hipMemsetAsync(d_within_range, true, sizeof(bool), stream);

    dim3 block(256);
    dim3 grid((size + 255) / 256);
    check_range<T><<<grid, block, 0, stream>>>(buffer, size, min, max, d_within_range);

    bool result;
    cudaD2Hcpy(&result, d_within_range, 1);
    return result;
}

template bool invokeCheckRange<int>(
    int const* buffer, const size_t size, int min, int max, bool* d_within_range, hipStream_t stream);

/*
 *  Determine the total workspace size based on a vector containing multiple variable sizes.
 */
size_t calcAlignedSize(std::vector<size_t> const& sizes, const size_t ALIGN_BYTES)
{
    const size_t ALIGN_MASK = ~(ALIGN_BYTES - 1);
    // Check ALIGN_BYTES is a power of 2
    assert((ALIGN_BYTES & (ALIGN_BYTES - 1)) == 0);

    size_t total = 0;
    for (auto sz : sizes)
    {
        total += (sz + ALIGN_BYTES - 1) & ALIGN_MASK;
    }

    // We add extra "ALIGN_BYTES - 1" bytes in case the start address passed to the function calcAlignedPointers() is
    // not aligned.
    return total + ALIGN_BYTES - 1;
}

/*
 * Given the address of the workspace and the vector containing multiple variable sizes, calculate the start addresses
 * of each variable.
 */
void calcAlignedPointers(
    std::vector<void*>& outPtrs, void const* p, std::vector<size_t> const& sizes, size_t ALIGN_BYTES)
{
    const size_t ALIGN_MASK = ~(ALIGN_BYTES - 1);
    // Check ALIGN_BYTES is a power of 2
    assert((ALIGN_BYTES & (ALIGN_BYTES - 1)) == 0);

    // In case the start address is not aligned
    char* ptr = reinterpret_cast<char*>((reinterpret_cast<size_t>(p) + ALIGN_BYTES - 1) & ALIGN_MASK);

    outPtrs.reserve(sizes.size());
    for (auto sz : sizes)
    {
        outPtrs.push_back(ptr);
        ptr += (sz + ALIGN_BYTES - 1) & ALIGN_MASK;
    }
}

} // namespace common
} // namespace tensorrt_llm
