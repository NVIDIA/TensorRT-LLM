#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaFp8Utils.h"
#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/envUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include <algorithm>
#include <cstdio>
#include <hip/hip_fp16.h>
#include <limits>
#include <type_traits>

namespace tensorrt_llm
{
namespace common
{
#ifdef ENABLE_FP8

constexpr int CTA_SIZE = 256;

template <bool QUANTIZE>
__inline__ __device__ float scale(float a, float b)
{
    return QUANTIZE ? a / b : a * b;
}

template <QuantizeMode QUANTIZE_MODE, bool QUANTIZE, typename T_OUT, typename T_S, typename T_IN>
__global__ void scaleMatrix(T_OUT* output, T_S const* input_scale, T_IN const* input, int64_t numel, int64_t lda)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.wait;");
#endif

    for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < numel; i += blockDim.x * gridDim.x)
    {

        if (QUANTIZE_MODE == QuantizeMode::PER_CHANNEL)
        {
            output[i] = T_OUT(scale<QUANTIZE>(static_cast<float>(input[i]), static_cast<float>(input_scale[i % lda])));
        }
        else if (QUANTIZE_MODE == QuantizeMode::PER_TOKEN)
        {
            output[i] = T_OUT(scale<QUANTIZE>(static_cast<float>(input[i]), static_cast<float>(input_scale[i / lda])));
        }
        else if (QUANTIZE_MODE == QuantizeMode::PER_TENSOR)
        {
            output[i] = T_OUT(scale<QUANTIZE>(static_cast<float>(input[i]), static_cast<float>(input_scale[0])));
        }
    }
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900))
    asm volatile("griddepcontrol.launch_dependents;");
#endif
}

template <typename T_OUT, typename T_S, typename T_IN>
void invokeQuantizeMatrix(T_OUT* output, T_S const* input_scale, T_IN const* input, int64_t numel, int64_t lda,
    QuantizeMode quantize_mode, hipStream_t stream)
{
    dim3 grid(1024);
    dim3 block(CTA_SIZE);
    hipLaunchConfig_t config;
    config.gridDim = grid;
    config.blockDim = block;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    if (quantize_mode == QuantizeMode::PER_CHANNEL)
    {
        cudaLaunchKernelEx(&config, scaleMatrix<QuantizeMode::PER_CHANNEL, true, T_OUT, T_S, T_IN>, output, input_scale,
            input, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TOKEN)
    {
        cudaLaunchKernelEx(&config, scaleMatrix<QuantizeMode::PER_TOKEN, true, T_OUT, T_S, T_IN>, output, input_scale,
            input, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TENSOR)
    {
        cudaLaunchKernelEx(&config, scaleMatrix<QuantizeMode::PER_TENSOR, true, T_OUT, T_S, T_IN>, output, input_scale,
            input, numel, lda);
    }
    sync_check_cuda_error(stream);
}

template <typename T_OUT, typename T_S, typename T_IN>
void invokeDequantizeMatrix(T_OUT* output, T_S const* input_scale, T_IN const* input, int64_t numel, int64_t lda,
    QuantizeMode quantize_mode, hipStream_t stream)
{
    dim3 grid(1024);
    dim3 block(CTA_SIZE);
    hipLaunchConfig_t config;
    config.gridDim = grid;
    config.blockDim = block;
    config.dynamicSmemBytes = 0;
    config.stream = stream;
    hipLaunchAttribute attrs[1];
    attrs[0].id = cudaLaunchAttributeProgrammaticStreamSerialization;
    attrs[0].val.programmaticStreamSerializationAllowed = tensorrt_llm::common::getEnvEnablePDL();
    config.numAttrs = 1;
    config.attrs = attrs;
    if (quantize_mode == QuantizeMode::PER_CHANNEL)
    {
        cudaLaunchKernelEx(&config, scaleMatrix<QuantizeMode::PER_CHANNEL, false, T_OUT, T_S, T_IN>, output,
            input_scale, input, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TOKEN)
    {
        cudaLaunchKernelEx(&config, scaleMatrix<QuantizeMode::PER_TOKEN, false, T_OUT, T_S, T_IN>, output, input_scale,
            input, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TENSOR)
    {
        cudaLaunchKernelEx(&config, scaleMatrix<QuantizeMode::PER_TENSOR, false, T_OUT, T_S, T_IN>, output, input_scale,
            input, numel, lda);
    }
    sync_check_cuda_error(stream);
}

template <typename T_FAKE, typename T_OUT, typename T_IN>
__global__ void fakeQuantize(T_OUT* dst, const T_IN* src, const int64_t numel)
{
    for (int64_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < numel; tid += blockDim.x * gridDim.x)
    {
        T_FAKE tmp = (T_FAKE) (static_cast<float>(src[tid]));
        dst[tid] = (T_OUT) (static_cast<float>(tmp));
    }
}

template <typename T_FAKE, typename T_OUT, typename T_IN>
void invokeFakeQuantize(T_OUT* dst, const T_IN* src, const int64_t numel, hipStream_t stream)
{
    fakeQuantize<T_FAKE><<<1024, CTA_SIZE, 0, stream>>>(dst, src, numel);
    sync_check_cuda_error(stream);
}

template void invokeFakeQuantize<__hip_fp8_e4m3_fnuz, float, float>(
    float* dst, float const* src, const int64_t numel, hipStream_t stream);
template void invokeFakeQuantize<float, float, __hip_fp8_e4m3_fnuz>(
    float* dst, __hip_fp8_e4m3_fnuz const* src, const int64_t numel, hipStream_t stream);
template void invokeFakeQuantize<__hip_fp8_e4m3_fnuz, half, half>(
    half* dst, half const* src, const int64_t numel, hipStream_t stream);
template void invokeFakeQuantize<__hip_fp8_e4m3_fnuz, __hip_bfloat16, __hip_bfloat16>(
    __hip_bfloat16* dst, __hip_bfloat16 const* src, const int64_t numel, hipStream_t stream);

template void invokeFakeQuantize<float, half, float>(
    half* dst, float const* src, const int64_t numel, hipStream_t stream);

__device__ float atomicMaxExtd(float* address, float val)
{
    assert(val >= 0);
    unsigned int* address_as_u = reinterpret_cast<unsigned int*>(address);
    unsigned int old = atomicMax(address_as_u, __float_as_uint(val));
    return __uint_as_float(old);
}

template <typename T>
inline __device__ T atomicMaxExtdV2(T* address, T val)
{
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
    static_assert(std::is_same_v<T, half> | std::is_same_v<T, __hip_bfloat16>, "T needs to be either half or bfloat16");
    // The address in 64 bits.
    uint64_t address_u64 = reinterpret_cast<uint64_t const&>(address);

    // Pack the input value into 32 bits.
    union
    {
        T v[2];
        uint16_t u[2];
    } old, tmp = {};

    int const loc = (address_u64 & 0x2) >> 1;
    tmp.v[loc] = val;

    // 4B aligned pointer.
    auto aligned_address = reinterpret_cast<T*>(address_u64 & ~0x3ull);

    if constexpr (std::is_same_v<T, half>)
    {
        asm volatile("atom.global.v2.f16.max.noftz {%0, %1}, [%2], {%3, %4};"
                     : "=h"(old.u[0]), "=h"(old.u[1])
                     : "l"(aligned_address), "h"(tmp.u[0]), "h"(tmp.u[1]));
    }
    if constexpr (std::is_same_v<T, __hip_bfloat16>)
    {
        asm volatile("atom.global.v2.bf16.max.noftz {%0, %1}, [%2], {%3, %4};"
                     : "=h"(old.u[0]), "=h"(old.u[1])
                     : "l"(aligned_address), "h"(tmp.u[0]), "h"(tmp.u[1]));
    }

    // Return the correct half.
    return old.v[loc];
#endif
}

__device__ half atomicMaxExtd(half* address, half val)
{
    unsigned short int* address_as_u = reinterpret_cast<unsigned short int*>(address);
    unsigned short int old = *address_as_u, assumed;

    while (val > __ushort_as_half(old))
    {
        assumed = old;
        old = atomicCAS(address_as_u, assumed, __half_as_ushort(val));
    }

    return __ushort_as_half(old);
}

__device__ __hip_bfloat16 atomicMaxExtd(__hip_bfloat16* address, __hip_bfloat16 val)
{
#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 800))
    unsigned short int* address_as_u = reinterpret_cast<unsigned short int*>(address);
    unsigned short int old = *address_as_u, assumed;

    while (val > __ushort_as_bfloat16(old))
    {
        assumed = old;
        old = atomicCAS(address_as_u, assumed, __bfloat16_as_ushort(val));
    }

    return __ushort_as_bfloat16(old);
#else
    assert(0);
    asm volatile("brkpt;\n" ::);
    return __hip_bfloat16(0);
#endif
}

template <QuantizeMode QUANTIZE_MODE, typename T_S, typename T_W>
__global__ void computeFP8QuantizeScale(T_S* quant_ptr, const T_W* weights, const int64_t size, const int64_t n)
{
    constexpr float min_scaling_factor = 1.0f / (FP8_E4M3_MAX * 512.f);
    if (QUANTIZE_MODE == QuantizeMode::PER_CHANNEL)
    {
        for (int64_t col = threadIdx.x; col < n; col += blockDim.x)
        {
            float max = 0.f;
            for (int64_t i = col + n * blockIdx.x; i < size; i += gridDim.x * n)
            {
                auto val = fabs(static_cast<float>(weights[i]));
                max = max > val ? max : val;
            }
            auto const scale = (T_S) std::max(max / FP8_E4M3_MAX, min_scaling_factor);
#if defined(__CUDA_ARCH__) && (__CUDA_ARCH__ >= 900)
            if constexpr (std::is_same_v<T_S, float>)
            {
                atomicMaxExtd(quant_ptr + col, scale);
            }
            else
            {
                auto const address_u64 = reinterpret_cast<uint64_t>(quant_ptr + col);
                if ((col == 0 && address_u64 % 4 != 0) || (col == n - 1 && address_u64 % 4 == 0))
                    atomicMaxExtd(quant_ptr + col, scale);
                else
                    atomicMaxExtdV2(quant_ptr + col, scale);
            }
#else // Vector atomics require __CUDA_ARCH__ >= 900
            atomicMaxExtd(quant_ptr + col, scale);
#endif
        }
    }
    else if (QUANTIZE_MODE == QuantizeMode::PER_TOKEN)
    {
        auto const nrows = size / n;
        for (int64_t row = blockIdx.x; row < nrows; row += gridDim.x)
        {
            float max = 0.f;
            for (int64_t i = threadIdx.x; i < n; i += blockDim.x)
            {
                auto val = fabs(static_cast<float>(weights[row * n + i]));
                max = max > val ? max : val;
            }
            max = blockReduceMax<float>(max);
            if (threadIdx.x == 0)
            {
                auto const scale = (T_S) std::max(max / FP8_E4M3_MAX, min_scaling_factor);
                quant_ptr[row] = scale;
            }
        }
    }
    else if (QUANTIZE_MODE == QuantizeMode::PER_TENSOR)
    {
        float max = 0.f;
        for (int64_t i = threadIdx.x + blockIdx.x * blockDim.x; i < size; i += gridDim.x * blockDim.x)
        {
            auto val = fabs(static_cast<float>(weights[i]));
            max = max > val ? max : val;
        }
        max = blockReduceMax<float>(max);
        if (threadIdx.x == 0)
        {
            auto const scale = (T_S) std::max(max / FP8_E4M3_MAX, min_scaling_factor);
            atomicMaxExtd(quant_ptr, scale);
        }
    }
}

template <typename T_S, typename T_W>
void invokeComputeFP8QuantizeScale(T_S* quant_ptr, const T_W* weights, const int64_t numel, const int64_t lda,
    QuantizeMode quantize_mode, hipStream_t stream)
{
    if (quantize_mode == QuantizeMode::PER_TOKEN)
    {
        dim3 block(CTA_SIZE);
        dim3 grid(numel / lda);
        computeFP8QuantizeScale<QuantizeMode::PER_TOKEN><<<grid, block, 0, stream>>>(quant_ptr, weights, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_CHANNEL)
    {
        dim3 block(CTA_SIZE);
        dim3 grid((lda + CTA_SIZE - 1) / CTA_SIZE);
        hipMemsetAsync(quant_ptr, 0, lda * sizeof(T_S), stream);
        sync_check_cuda_error(stream);
        computeFP8QuantizeScale<QuantizeMode::PER_CHANNEL><<<grid, block, 0, stream>>>(quant_ptr, weights, numel, lda);
    }
    else if (quantize_mode == QuantizeMode::PER_TENSOR)
    {
        dim3 block(1024);
        dim3 grid(1024);
        hipMemsetAsync(quant_ptr, 0, sizeof(T_S), stream);
        sync_check_cuda_error(stream);
        computeFP8QuantizeScale<QuantizeMode::PER_TENSOR><<<grid, block, 0, stream>>>(quant_ptr, weights, numel, lda);
    }
    sync_check_cuda_error(stream);
}

#define DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(type_scale, type_in)                                                  \
    template void invokeComputeFP8QuantizeScale<type_scale, type_in>(type_scale * input_scale, type_in const* weights, \
        int64_t numel, int64_t lda, QuantizeMode quantize_mode, hipStream_t stream);

DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(half, half);
DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(float, half);
DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(float, float);
#ifdef ENABLE_BF16
DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(__hip_bfloat16, __hip_bfloat16);
DEFINE_INVOKE_COMPUTE_FP8_QUANTIZE_SCALE(float, __hip_bfloat16);
#endif

template <typename T_OUT, typename T_S, typename T_IN>
__global__ void dynamicQuantizeMatrixPerToken(
    T_OUT* output, T_S* quant_ptr, T_IN const* input, int64_t numel, int64_t lda)
{
    extern __shared__ __align__(sizeof(float)) char _shmem[];
    T_IN* shmem = reinterpret_cast<T_IN*>(_shmem);
    constexpr float min_scaling_factor = 1.0f / (FP8_E4M3_MAX * 512.f);
    auto const nrows = numel / lda;
    for (int64_t row = blockIdx.x; row < nrows; row += gridDim.x)
    {
        float max = 0.f;
        for (int64_t i = threadIdx.x; i < lda; i += blockDim.x)
        {
            auto const in = input[row * lda + i];
            shmem[i] = in;
            auto val = fabs(static_cast<float>(in));
            max = max > val ? max : val;
        }
        max = blockAllReduceMax<float>(max); // __syncthreads() called so we can read shmem
        auto const s = (T_S) std::max(max / FP8_E4M3_MAX, min_scaling_factor);
        for (int64_t i = threadIdx.x; i < lda; i += blockDim.x)
        {
            // true means we are quantizing
            output[row * lda + i] = (T_OUT) scale<true>(static_cast<float>(shmem[i]), static_cast<float>(s));
        }
        if (threadIdx.x == 0)
        {
            quant_ptr[row] = s;
        }
    }
}

template <typename T_OUT, typename T_S, typename T_IN>
void invokeComputeScalesAndQuantizeMatrix(T_OUT* output, T_S* quant_ptr, const T_IN* input, const int64_t numel,
    const int64_t lda, QuantizeMode quantize_mode, hipStream_t stream)
{
    if (quantize_mode == QuantizeMode::PER_TOKEN)
    {
        dim3 grid(numel / lda);
        bool use_shmem = true;
        auto const shmem_size = lda * sizeof(T_IN);
        if (shmem_size >= (48 << 10))
        {
            hipError_t ret = hipFuncSetAttribute(reinterpret_cast<const void*>(dynamicQuantizeMatrixPerToken<T_OUT), T_S, T_IN>,
                hipFuncAttributeMaxDynamicSharedMemorySize, shmem_size);
            use_shmem = ret == hipSuccess;
        }
        if (use_shmem)
        {
            // ensure the threadblock is as large as possible to increase occupancy
            dim3 block(std::min((lda + 31) / 32 * 32, static_cast<int64_t>(1024)));
            dynamicQuantizeMatrixPerToken<<<grid, block, shmem_size, stream>>>(output, quant_ptr, input, numel, lda);
        }
        else
        {
            dim3 block(CTA_SIZE);
            computeFP8QuantizeScale<QuantizeMode::PER_TOKEN><<<grid, block, 0, stream>>>(quant_ptr, input, numel, lda);
            sync_check_cuda_error(stream);
            invokeQuantizeMatrix(output, quant_ptr, input, numel, lda, quantize_mode, stream);
        }
    }
    else if (quantize_mode == QuantizeMode::PER_CHANNEL)
    {
        dim3 block(CTA_SIZE);
        dim3 grid((lda + CTA_SIZE - 1) / CTA_SIZE);
        hipMemsetAsync(quant_ptr, 0, lda * sizeof(T_S), stream);
        sync_check_cuda_error(stream);
        computeFP8QuantizeScale<QuantizeMode::PER_CHANNEL><<<grid, block, 0, stream>>>(quant_ptr, input, numel, lda);
        sync_check_cuda_error(stream);
        invokeQuantizeMatrix(output, quant_ptr, input, numel, lda, quantize_mode, stream);
    }
    else if (quantize_mode == QuantizeMode::PER_TENSOR)
    {
        dim3 block(1024);
        dim3 grid(1024);
        hipMemsetAsync(quant_ptr, 0, sizeof(T_S), stream);
        sync_check_cuda_error(stream);
        computeFP8QuantizeScale<QuantizeMode::PER_TENSOR><<<grid, block, 0, stream>>>(quant_ptr, input, numel, lda);
        sync_check_cuda_error(stream);
        invokeQuantizeMatrix(output, quant_ptr, input, numel, lda, quantize_mode, stream);
    }
    sync_check_cuda_error(stream);
}

#define DEFINE_INVOKE_QUANTIZE_MATRIX(type_out, type_scale, type_in)                                                   \
    template void invokeQuantizeMatrix<type_out, type_scale, type_in>(type_out * output,                               \
        type_scale const* input_scale, type_in const* input, int64_t numel, int64_t lda, QuantizeMode quantize_mode,   \
        hipStream_t stream);                                                                                          \
    template void invokeDequantizeMatrix<type_out, type_scale, type_in>(type_out * output,                             \
        type_scale const* input_scale, type_in const* input, int64_t numel, int64_t lda, QuantizeMode quantize_mode,   \
        hipStream_t stream);                                                                                          \
    template void invokeComputeScalesAndQuantizeMatrix<type_out, type_scale, type_in>(type_out * output,               \
        type_scale * input_scale, type_in const* input, int64_t numel, int64_t lda, QuantizeMode quantize_mode,        \
        hipStream_t stream);

#ifdef ENABLE_FP8
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, float, float);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, float, half);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, half, half);
DEFINE_INVOKE_QUANTIZE_MATRIX(half, half, __hip_fp8_e4m3_fnuz);
DEFINE_INVOKE_QUANTIZE_MATRIX(float, float, __hip_fp8_e4m3_fnuz);
DEFINE_INVOKE_QUANTIZE_MATRIX(half, float, __hip_fp8_e4m3_fnuz);
#ifdef ENABLE_BF16
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, float, __hip_bfloat16);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_fp8_e4m3_fnuz, __hip_bfloat16, __hip_bfloat16);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_bfloat16, __hip_bfloat16, __hip_fp8_e4m3_fnuz);
DEFINE_INVOKE_QUANTIZE_MATRIX(__hip_bfloat16, float, __hip_fp8_e4m3_fnuz);
#endif
#endif

#endif // ENABLE_FP8
} // namespace common
} // namespace tensorrt_llm
