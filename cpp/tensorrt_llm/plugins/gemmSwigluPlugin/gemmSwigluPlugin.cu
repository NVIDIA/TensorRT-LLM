/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2023 NVIDIA CORPORATION &
 * AFFILIATES. All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "gemmSwigluPlugin.h"

#include "cutlass/util/reference/device/tensor_fill.h"
#include "cutlass_extensions/gemm_configs.h"

using namespace nvinfer1;
using namespace tensorrt_llm::common;
using namespace tensorrt_llm::kernels::cutlass_kernels;
using tensorrt_llm::plugins::GemmSwigluPluginCreator;
using tensorrt_llm::plugins::GemmSwigluPlugin;
using tensorrt_llm::plugins::GemmSwigluPluginProfiler;
using tensorrt_llm::plugins::read;
using tensorrt_llm::plugins::write;

void GemmSwigluPluginProfiler::initTmpData(int m, int n, int k, char* workspace, size_t size, hipStream_t stream)
{
    size_t bpe = getBytePerElement(mType);

    if (mType == nvinfer1::DataType::kFP8)
    {
        cutlass::reference::device::BlockFillRandomUniform(reinterpret_cast<cutlass::float_e4m3_t*>(workspace),
            m * k + n * k + 1 * n, 42, cutlass::float_e4m3_t{128}, -cutlass::float_e4m3_t{128}, -1, 0, stream);
    }
}
